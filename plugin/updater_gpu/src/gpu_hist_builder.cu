#include "hip/hip_runtime.h"
/*!
 * Copyright 2017 Rory mitchell
*/
#include <thrust/binary_search.h>
#include <thrust/count.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <hipcub/hipcub.hpp>
#include <numeric>
#include <algorithm>
#include <functional>
#include "common.cuh"
#include "device_helpers.cuh"
#include "gpu_hist_builder.cuh"

namespace xgboost {
namespace tree {
void DeviceGMat::Init(const common::GHistIndexMatrix& gmat) {
  CHECK_EQ(gidx.size(), gmat.index.size())
      << "gidx must be externally allocated";
  CHECK_EQ(ridx.size(), gmat.index.size())
      << "ridx must be externally allocated";

  gidx = gmat.index;
  thrust::device_vector<int> row_ptr = gmat.row_ptr;

  auto counting = thrust::make_counting_iterator(0);
  thrust::upper_bound(row_ptr.begin(), row_ptr.end(), counting,
                      counting + gidx.size(), ridx.tbegin());
  thrust::transform(ridx.tbegin(), ridx.tend(), ridx.tbegin(),
                    [=] __device__(int val) { return val - 1; });
}

void DeviceHist::Init(int n_bins_in) {
  this->n_bins = n_bins_in;
  CHECK(!hist.empty()) << "DeviceHist must be externally allocated";
}

void DeviceHist::Reset() { hist.fill(gpu_gpair()); }

gpu_gpair* DeviceHist::GetLevelPtr(int depth) {
  return hist.data() + n_nodes(depth - 1) * n_bins;
}

int DeviceHist::LevelSize(int depth) { return n_bins * n_nodes_level(depth); }

HistBuilder DeviceHist::GetBuilder() {
  return HistBuilder(hist.data(), n_bins);
}

HistBuilder::HistBuilder(gpu_gpair* ptr, int n_bins)
    : d_hist(ptr), n_bins(n_bins) {}

__device__ void HistBuilder::Add(gpu_gpair gpair, int gidx, int nidx) const {
  int hist_idx = nidx * n_bins + gidx;
  atomicAdd(&(d_hist[hist_idx]._grad), gpair._grad);
  atomicAdd(&(d_hist[hist_idx]._hess), gpair._hess);
}

__device__ gpu_gpair HistBuilder::Get(int gidx, int nidx) const {
  return d_hist[nidx * n_bins + gidx];
}

GPUHistBuilder::GPUHistBuilder() {}

GPUHistBuilder::~GPUHistBuilder() {}

void GPUHistBuilder::Init(const TrainParam& param) {
  CHECK(param.max_depth < 16) << "Tree depth too large.";
  this->param = param;
  initialised = false;
  is_dense = false;

  dh::safe_cuda(hipSetDevice(param.gpu_id));
  if (!param.silent) {
    LOG(CONSOLE) << "Device: [" << param.gpu_id << "] " << dh::device_name();
  }
}

template <typename ReductionOpT>
struct ReduceBySegmentOp {
  ReductionOpT op;

  __host__ __device__ __forceinline__ ReduceBySegmentOp() {}

  __host__ __device__ __forceinline__ ReduceBySegmentOp(ReductionOpT op)
      : op(op) {}

  template <typename KeyValuePairT>
  __host__ __device__ __forceinline__ KeyValuePairT
  operator()(const KeyValuePairT& first, const KeyValuePairT& second) {
    KeyValuePairT retval;
    retval.key = second.key;
    retval.value =
        first.key != second.key ? second.value : op(first.value, second.value);
    return retval;
  }
};

template <int ITEMS_PER_THREAD, int BLOCK_THREADS>
__global__ void hist_kernel(gpu_gpair* d_dense_hist, int* d_ridx, int* d_gidx,
                            NodeIdT* d_position, gpu_gpair* d_gpair, int n_bins,
                            int depth, int n) {
  typedef hipcub::KeyValuePair<int, gpu_gpair> OffsetValuePairT;
  typedef hipcub::BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD,
                         hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoadT;
  typedef hipcub::BlockRadixSort<int, BLOCK_THREADS, ITEMS_PER_THREAD, int>
      BlockRadixSortT;
  typedef hipcub::BlockDiscontinuity<int, BLOCK_THREADS> BlockDiscontinuityKeysT;
  typedef ReduceBySegmentOp<hipcub::Sum> ReduceBySegmentOpT;
  typedef hipcub::BlockScan<OffsetValuePairT, BLOCK_THREADS,
                         hipcub::BLOCK_SCAN_WARP_SCANS>
      BlockScanT;

  union TempStorage {
    typename BlockLoadT::TempStorage load;
    typename BlockRadixSortT::TempStorage sort;
    typename BlockScanT::TempStorage scan;
    typename BlockDiscontinuityKeysT::TempStorage disc;
  };

  __shared__ TempStorage temp_storage;

  int ridx[ITEMS_PER_THREAD];
  int gidx[ITEMS_PER_THREAD];

  const int TILE_SIZE = ITEMS_PER_THREAD * BLOCK_THREADS;
  int block_offset = blockIdx.x * TILE_SIZE;

  BlockLoadT(temp_storage.load)
      .Load(d_ridx + block_offset, ridx, n - block_offset, -1);
  BlockLoadT(temp_storage.load)
      .Load(d_gidx + block_offset, gidx, n - block_offset, -1);

  int hist_idx[ITEMS_PER_THREAD];

  for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++) {
    if (ridx[ITEM] > -1 && d_position[ridx[ITEM]] > -1) {
      hist_idx[ITEM] =
          (d_position[ridx[ITEM]] - n_nodes(depth - 1)) * n_bins + gidx[ITEM];
    } else {
      hist_idx[ITEM] = -1;
    }
  }

  __syncthreads();
  BlockRadixSortT(temp_storage.sort).Sort(hist_idx, ridx);

  OffsetValuePairT kv[ITEMS_PER_THREAD];

  for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++) {
    kv[ITEM].key = hist_idx[ITEM];
    if (ridx[ITEM] > -1) {
      kv[ITEM].value = d_gpair[ridx[ITEM]];
    }
  }

  __syncthreads();
  // Scan
  BlockScanT(temp_storage.scan).InclusiveScan(kv, kv, ReduceBySegmentOpT());

  __syncthreads();
  int flags[ITEMS_PER_THREAD];
  BlockDiscontinuityKeysT(temp_storage.disc)
      .FlagTails(flags, hist_idx, hipcub::Inequality());

  for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++) {
    if (flags[ITEM]) {
      if (ridx[ITEM] > -1 && d_position[ridx[ITEM]] > -1) {
        atomicAdd(&(d_dense_hist[hist_idx[ITEM]]._grad), kv[ITEM].value._grad);
        atomicAdd(&(d_dense_hist[hist_idx[ITEM]]._hess), kv[ITEM].value._hess);
      }
    }
  }
}

void GPUHistBuilder::BuildHist(int depth) {
  auto d_ridx = device_matrix.ridx.data();
  auto d_gidx = device_matrix.gidx.data();
  auto d_position = position.data();
  auto d_gpair = device_gpair.data();
  auto hist_builder = hist.GetBuilder();

  dh::launch_n(device_matrix.gidx.size(), [=] __device__(int idx) {
    int ridx = d_ridx[idx];
    int pos = d_position[ridx];

    // Only increment even nodes
    if (pos < 0 || pos % 2 == 1) return;

    int gidx = d_gidx[idx];
    gpu_gpair gpair = d_gpair[ridx];

    hist_builder.Add(gpair, gidx, pos);
  });

  dh::safe_cuda(hipDeviceSynchronize());

  // Subtraction trick
  int n_sub_bins = (n_nodes_level(depth) / 2) * hist_builder.n_bins;
  if (n_sub_bins > 0) {
    dh::launch_n(n_sub_bins, [=] __device__(int idx) {
      int nidx = n_nodes(depth - 1) + ((idx / hist_builder.n_bins) * 2);
      int gidx = idx % hist_builder.n_bins;
      gpu_gpair parent = hist_builder.Get(gidx, nidx / 2);
      gpu_gpair right = hist_builder.Get(gidx, nidx + 1);
      hist_builder.Add(parent - right, gidx, nidx);
    });
  }
  dh::safe_cuda(hipDeviceSynchronize());
}

void GPUHistBuilder::FindSplit(int depth) {
  auto counting = thrust::make_counting_iterator(0);
  auto d_gidx_feature_map = gidx_feature_map.data();
  int n_bins = hmat_.row_ptr.back();
  int n_features = hmat_.row_ptr.size() - 1;

  auto feature_boundary = [=] __device__(int idx_a, int idx_b) {
    int gidx_a = idx_a % n_bins;
    int gidx_b = idx_b % n_bins;
    return d_gidx_feature_map[gidx_a] == d_gidx_feature_map[gidx_b];
  };  // NOLINT

  // Reduce node sums
  {
    size_t temp_storage_bytes;
    hipcub::DeviceSegmentedReduce::Reduce(
        nullptr, temp_storage_bytes, hist.GetLevelPtr(depth), node_sums.data(),
        n_nodes_level(depth) * n_features, feature_segments.data(),
        feature_segments.data() + 1, hipcub::Sum(), gpu_gpair());
    cub_mem.LazyAllocate(temp_storage_bytes);
    hipcub::DeviceSegmentedReduce::Reduce(
        cub_mem.d_temp_storage, cub_mem.temp_storage_bytes,
        hist.GetLevelPtr(depth), node_sums.data(),
        n_nodes_level(depth) * n_features, feature_segments.data(),
        feature_segments.data() + 1, hipcub::Sum(), gpu_gpair());
  }

  // Scan
  thrust::exclusive_scan_by_key(
      counting, counting + hist.LevelSize(depth),
      thrust::device_pointer_cast(hist.GetLevelPtr(depth)), hist_scan.tbegin(),
      gpu_gpair(), feature_boundary);

  // Calculate gain
  auto d_gain = gain.data();
  auto d_nodes = nodes.data();
  auto d_node_sums = node_sums.data();
  auto d_hist_scan = hist_scan.data();
  GPUTrainingParam gpu_param_alias =
      gpu_param;  // Must be local variable to be used in device lambda
  bool colsample =
      param.colsample_bylevel < 1.0 || param.colsample_bytree < 1.0;
  auto d_feature_flags = feature_flags.data();

  dh::launch_n(hist.LevelSize(depth), [=] __device__(int idx) {
    int node_segment = idx / n_bins;
    int node_idx = n_nodes(depth - 1) + node_segment;
    gpu_gpair parent_sum = d_nodes[node_idx].sum_gradients;
    float parent_gain = d_nodes[node_idx].root_gain;
    int gidx = idx % n_bins;
    int findex = d_gidx_feature_map[gidx];

    // colsample
    if (colsample && d_feature_flags[d_gidx_feature_map[gidx]] == 0) {
      d_gain[idx] = 0;
    } else {
      gpu_gpair scan = d_hist_scan[idx];
      gpu_gpair sum = d_node_sums[node_segment * n_features + findex];
      gpu_gpair missing = parent_sum - sum;

      bool missing_left;
      d_gain[idx] = loss_chg_missing(scan, missing, parent_sum, parent_gain,
                                     gpu_param_alias, missing_left);
    }
  });
  dh::safe_cuda(hipDeviceSynchronize());

  // Find best gain
  {
    size_t temp_storage_bytes;
    hipcub::DeviceSegmentedReduce::ArgMax(nullptr, temp_storage_bytes, gain.data(),
                                       argmax.data(), n_nodes_level(depth),
                                       hist_node_segments.data(),
                                       hist_node_segments.data() + 1);
    cub_mem.LazyAllocate(temp_storage_bytes);
    hipcub::DeviceSegmentedReduce::ArgMax(
        cub_mem.d_temp_storage, cub_mem.temp_storage_bytes, gain.data(),
        argmax.data(), n_nodes_level(depth), hist_node_segments.data(),
        hist_node_segments.data() + 1);
  }

  auto d_argmax = argmax.data();
  auto d_gidx_fvalue_map = gidx_fvalue_map.data();
  auto d_fidx_min_map = fidx_min_map.data();
  dh::launch_n(n_nodes_level(depth), [=] __device__(int idx) {
    int max_idx = n_bins * idx + d_argmax[idx].key;
    int gidx = max_idx % n_bins;
    int fidx = d_gidx_feature_map[gidx];
    int node_segment = max_idx / n_bins;
    int node_idx = n_nodes(depth - 1) + node_segment;
    gpu_gpair scan = d_hist_scan[max_idx];
    gpu_gpair parent_sum = d_nodes[node_idx].sum_gradients;
    float parent_gain = d_nodes[node_idx].root_gain;
    gpu_gpair sum = d_node_sums[node_segment * n_features + fidx];
    gpu_gpair missing = parent_sum - sum;

    bool missing_left;
    float loss_chg = loss_chg_missing(scan, missing, parent_sum, parent_gain,
                                      gpu_param_alias, missing_left);

    float fvalue;
    if (gidx == 0 || fidx != d_gidx_feature_map[gidx - 1]) {
      fvalue = d_fidx_min_map[fidx];
    } else {
      fvalue = d_gidx_fvalue_map[gidx - 1];
    }

    gpu_gpair left = missing_left ? scan + missing : scan;
    gpu_gpair right = parent_sum - left;
    d_nodes[node_idx].split.Update(loss_chg, missing_left, fvalue, fidx, left,
                                   right, gpu_param_alias);

    int left_child_idx = n_nodes(depth) + idx * 2;
    int right_child_idx = n_nodes(depth) + idx * 2 + 1;
    d_nodes[left_child_idx] =
        Node(left, CalcGain(gpu_param_alias, left.grad(), left.hess()),
             CalcWeight(gpu_param_alias, left.grad(), left.hess()));

    d_nodes[right_child_idx] =
        Node(right, CalcGain(gpu_param_alias, right.grad(), right.hess()),
             CalcWeight(gpu_param_alias, right.grad(), right.hess()));
  });
  dh::safe_cuda(hipDeviceSynchronize());
}

void GPUHistBuilder::InitFirstNode() {
  // Build the root node on the CPU and copy to device
  gpu_gpair sum_gradients =
      thrust::reduce(device_gpair.tbegin(), device_gpair.tend(),
                     gpu_gpair(0, 0), thrust::plus<gpu_gpair>());

  Node tmp =
      Node(sum_gradients,
           CalcGain(param, sum_gradients.grad(), sum_gradients.hess()),
           CalcWeight(param, sum_gradients.grad(), sum_gradients.hess()));

  thrust::copy_n(&tmp, 1, nodes.tbegin());
}

void GPUHistBuilder::UpdatePosition() {
  if (is_dense) {
    this->UpdatePositionDense();
  } else {
    this->UpdatePositionSparse();
  }
}

void GPUHistBuilder::UpdatePositionDense() {
  auto d_position = position.data();
  Node* d_nodes = nodes.data();
  auto d_gidx_fvalue_map = gidx_fvalue_map.data();
  auto d_gidx = device_matrix.gidx.data();
  int n_columns = info->num_col;

  dh::launch_n(position.size(), [=] __device__(int idx) {
    NodeIdT pos = d_position[idx];
    if (pos < 0) {
      return;
    }

    Node node = d_nodes[pos];

    if (node.IsLeaf()) {
      d_position[idx] = -1;
      return;
    }

    int gidx = d_gidx[idx * n_columns + node.split.findex];

    float fvalue = d_gidx_fvalue_map[gidx];

    if (fvalue <= node.split.fvalue) {
      d_position[idx] = pos * 2 + 1;
    } else {
      d_position[idx] = pos * 2 + 2;
    }
  });
}

void GPUHistBuilder::UpdatePositionSparse() {
  auto d_position = position.data();
  auto d_position_tmp = position_tmp.data();
  Node* d_nodes = nodes.data();
  auto d_gidx_feature_map = gidx_feature_map.data();
  auto d_gidx_fvalue_map = gidx_fvalue_map.data();
  auto d_gidx = device_matrix.gidx.data();
  auto d_ridx = device_matrix.ridx.data();

  // Update missing direction
  dh::launch_n(position.size(), [=] __device__(int idx) {
    NodeIdT pos = d_position[idx];
    if (pos < 0) {
      return;
    }

    Node node = d_nodes[pos];

    if (node.IsLeaf()) {
      d_position_tmp[idx] = -1;
    } else if (node.split.missing_left) {
      d_position_tmp[idx] = pos * 2 + 1;
    } else {
      d_position_tmp[idx] = pos * 2 + 2;
    }
  });

  // Update node based on fvalue where exists
  dh::launch_n(device_matrix.gidx.size(), [=] __device__(int idx) {
    int ridx = d_ridx[idx];
    NodeIdT pos = d_position[ridx];
    if (pos < 0) {
      return;
    }

    Node node = d_nodes[pos];

    if (node.IsLeaf()) {
      return;
    }

    int gidx = d_gidx[idx];
    int findex = d_gidx_feature_map[gidx];

    if (findex == node.split.findex) {
      float fvalue = d_gidx_fvalue_map[gidx];

      if (fvalue <= node.split.fvalue) {
        d_position_tmp[ridx] = pos * 2 + 1;
      } else {
        d_position_tmp[ridx] = pos * 2 + 2;
      }
    }
  });

  position = position_tmp;
}

void GPUHistBuilder::ColSampleTree() {
  feature_set_tree.resize(info->num_col);
  std::iota(feature_set_tree.begin(), feature_set_tree.end(), 0);
  feature_set_tree = col_sample(feature_set_tree, param.colsample_bytree);
}

void GPUHistBuilder::ColSampleLevel() {
  feature_set_level.resize(feature_set_tree.size());
  feature_set_level = col_sample(feature_set_tree, param.colsample_bylevel);
  std::vector<int> h_feature_flags(info->num_col, 0);
  for (auto fidx : feature_set_level) {
    h_feature_flags[fidx] = 1;
  }
  feature_flags = h_feature_flags;
}

void GPUHistBuilder::InitData(const std::vector<bst_gpair>& gpair,
                              DMatrix& fmat,  // NOLINT
                              const RegTree& tree) {
  if (!initialised) {
    CHECK(fmat.SingleColBlock()) << "grow_gpu_hist: must have single column "
                                    "block. Try setting 'tree_method' "
                                    "parameter to 'exact'";
    info = &fmat.info();
    is_dense = info->num_nonzero == info->num_col * info->num_row;
    hmat_.Init(&fmat, param.max_bin);
    gmat_.cut = &hmat_;
    gmat_.Init(&fmat);
    int n_bins = hmat_.row_ptr.back();
    int n_features = hmat_.row_ptr.size() - 1;

    // Build feature segments
    std::vector<int> h_feature_segments;
    for (int node = 0; node < n_nodes_level(param.max_depth - 1); node++) {
      for (int fidx = 0; fidx < hmat_.row_ptr.size() - 1; fidx++) {
        h_feature_segments.push_back(hmat_.row_ptr[fidx] + node * n_bins);
      }
    }
    h_feature_segments.push_back(n_nodes_level(param.max_depth - 1) * n_bins);

    int level_max_bins = n_nodes_level(param.max_depth - 1) * n_bins;

    size_t free_memory = dh::available_memory();
    ba.allocate(
        &gidx_feature_map, n_bins, &hist_node_segments,
        n_nodes_level(param.max_depth - 1) + 1, &feature_segments,
        h_feature_segments.size(), &gain, level_max_bins, &position,
        gpair.size(), &position_tmp, gpair.size(), &nodes,
        n_nodes(param.max_depth), &gidx_fvalue_map, hmat_.cut.size(),
        &fidx_min_map, hmat_.min_val.size(), &argmax,
        n_nodes_level(param.max_depth - 1), &node_sums,
        n_nodes_level(param.max_depth - 1) * n_features, &hist_scan,
        level_max_bins, &device_gpair, gpair.size(), &device_matrix.gidx,
        gmat_.index.size(), &device_matrix.ridx, gmat_.index.size(), &hist.hist,
        n_nodes(param.max_depth - 1) * n_bins, &feature_flags, n_features);

    if (!param.silent) {
      const int mb_size = 1048576;
      LOG(CONSOLE) << "Allocated " << ba.size() / mb_size << "/"
                   << free_memory / mb_size << " MB on " << dh::device_name();
    }

    // Construct feature map
    std::vector<int> h_gidx_feature_map(n_bins);
    for (int row = 0; row < hmat_.row_ptr.size() - 1; row++) {
      for (int i = hmat_.row_ptr[row]; i < hmat_.row_ptr[row + 1]; i++) {
        h_gidx_feature_map[i] = row;
      }
    }

    gidx_feature_map = h_gidx_feature_map;

    // Construct device matrix
    device_matrix.Init(gmat_);

    gidx_fvalue_map = hmat_.cut;
    fidx_min_map = hmat_.min_val;

    thrust::sequence(hist_node_segments.tbegin(), hist_node_segments.tend(), 0,
                     n_bins);

    feature_segments = h_feature_segments;

    hist.Init(n_bins);

    initialised = true;
  }
  nodes.fill(Node());
  position.fill(0);
  device_gpair = gpair;
  subsample_gpair(&device_gpair, param.subsample);
  hist.Reset();
}

void GPUHistBuilder::Update(const std::vector<bst_gpair>& gpair,
                            DMatrix* p_fmat, RegTree* p_tree) {
  this->InitData(gpair, *p_fmat, *p_tree);
  this->InitFirstNode();
  this->ColSampleTree();

  for (int depth = 0; depth < param.max_depth; depth++) {
    this->ColSampleLevel();
    this->BuildHist(depth);
    this->FindSplit(depth);
    this->UpdatePosition();
  }
  dense2sparse_tree(p_tree, nodes.tbegin(), nodes.tend(), param);
}
}  // namespace tree
}  // namespace xgboost
