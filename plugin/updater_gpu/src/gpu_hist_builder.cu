#include "hip/hip_runtime.h"
/*!
 * Copyright 2017 Rory mitchell
 */
#include <thrust/binary_search.h>
#include <thrust/count.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <hipcub/hipcub.hpp>
#include <algorithm>
#include <functional>
#include <numeric>
#include "common.cuh"
#include "device_helpers.cuh"
#include "gpu_hist_builder.cuh"

namespace xgboost {
namespace tree {

void DeviceGMat::Init(const common::GHistIndexMatrix& gmat) {
  CHECK_EQ(gidx.size(), gmat.index.size())
      << "gidx must be externally allocated";
  CHECK_EQ(ridx.size(), gmat.index.size())
      << "ridx must be externally allocated";

  gidx = gmat.index;
  thrust::device_vector<int> row_ptr = gmat.row_ptr;

  auto counting = thrust::make_counting_iterator(0);
  thrust::upper_bound(row_ptr.begin(), row_ptr.end(), counting,
                      counting + gidx.size(), ridx.tbegin());
  thrust::transform(ridx.tbegin(), ridx.tend(), ridx.tbegin(),
                    [=] __device__(int val) { return val - 1; });
}

void DeviceHist::Init(int n_bins_in) {
  this->n_bins = n_bins_in;
  CHECK(!hist.empty()) << "DeviceHist must be externally allocated";
}

void DeviceHist::Reset() { hist.fill(gpu_gpair()); }

gpu_gpair* DeviceHist::GetLevelPtr(int depth) {
  return hist.data() + n_nodes(depth - 1) * n_bins;
}

int DeviceHist::LevelSize(int depth) { return n_bins * n_nodes_level(depth); }

HistBuilder DeviceHist::GetBuilder() {
  return HistBuilder(hist.data(), n_bins);
}

HistBuilder::HistBuilder(gpu_gpair* ptr, int n_bins)
    : d_hist(ptr), n_bins(n_bins) {}

__device__ void HistBuilder::Add(gpu_gpair gpair, int gidx, int nidx) const {
  int hist_idx = nidx * n_bins + gidx;
  atomicAdd(&(d_hist[hist_idx]._grad), gpair._grad);
  atomicAdd(&(d_hist[hist_idx]._hess), gpair._hess);
}

__device__ gpu_gpair HistBuilder::Get(int gidx, int nidx) const {
  return d_hist[nidx * n_bins + gidx];
}

GPUHistBuilder::GPUHistBuilder()
    : initialised(false),
      is_dense(false),
      p_last_fmat_(nullptr),
      prediction_cache_initialised(false) {}

GPUHistBuilder::~GPUHistBuilder() {}

void GPUHistBuilder::Init(const TrainParam& param) {
  CHECK(param.max_depth < 16) << "Tree depth too large.";
  CHECK(param.grow_policy != TrainParam::kLossGuide)
      << "Loss guided growth policy not supported. Use CPU algorithm.";
  this->param = param;

  dh::safe_cuda(hipSetDevice(param.gpu_id));
  if (!param.silent) {
    LOG(CONSOLE) << "Device: [" << param.gpu_id << "] " << dh::device_name();
  }
}

void GPUHistBuilder::BuildHist(int depth) {
  auto d_ridx = device_matrix.ridx.data();
  auto d_gidx = device_matrix.gidx.data();
  auto d_position = position.data();
  auto d_gpair = device_gpair.data();
  auto hist_builder = hist.GetBuilder();
  auto d_left_child_smallest = left_child_smallest.data();

  dh::launch_n(device_matrix.gidx.size(), [=] __device__(int idx) {
    int ridx = d_ridx[idx];
    int pos = d_position[ridx];
    if (!is_active(pos, depth)) return;

    // Only increment smallest node
    bool is_smallest =
        (d_left_child_smallest[parent_nidx(pos)] && is_left_child(pos)) ||
        (!d_left_child_smallest[parent_nidx(pos)] && !is_left_child(pos));
    if (!is_smallest && depth > 0) return;

    int gidx = d_gidx[idx];
    gpu_gpair gpair = d_gpair[ridx];

    hist_builder.Add(gpair, gidx, pos);
  });

  dh::safe_cuda(hipDeviceSynchronize());

  // Subtraction trick
  int n_sub_bins = (n_nodes_level(depth) / 2) * hist_builder.n_bins;
  if (depth > 0) {
    dh::launch_n(n_sub_bins, [=] __device__(int idx) {
      int nidx = n_nodes(depth - 1) + ((idx / hist_builder.n_bins) * 2);
      bool left_smallest = d_left_child_smallest[parent_nidx(nidx)];
      if (left_smallest) {
        nidx++;  // If left is smallest switch to right child
      }

      int gidx = idx % hist_builder.n_bins;
      gpu_gpair parent = hist_builder.Get(gidx, parent_nidx(nidx));
      int other_nidx = left_smallest ? nidx - 1 : nidx + 1;
      gpu_gpair other = hist_builder.Get(gidx, other_nidx);
      hist_builder.Add(parent - other, gidx, nidx);
    });
  }
  dh::safe_cuda(hipDeviceSynchronize());
}

template <int BLOCK_THREADS>
__global__ void find_split_kernel(
    const gpu_gpair* d_level_hist, int* d_feature_segments, int depth,
    int n_features, int n_bins, Node* d_nodes, float* d_fidx_min_map,
    float* d_gidx_fvalue_map, GPUTrainingParam gpu_param,
    bool* d_left_child_smallest, bool colsample, int* d_feature_flags) {
  typedef hipcub::KeyValuePair<int, float> ArgMaxT;
  typedef hipcub::BlockScan<gpu_gpair, BLOCK_THREADS, hipcub::BLOCK_SCAN_WARP_SCANS>
      BlockScanT;
  typedef hipcub::BlockReduce<ArgMaxT, BLOCK_THREADS> MaxReduceT;
  typedef hipcub::BlockReduce<gpu_gpair, BLOCK_THREADS> SumReduceT;

  union TempStorage {
    typename BlockScanT::TempStorage scan;
    typename MaxReduceT::TempStorage max_reduce;
    typename SumReduceT::TempStorage sum_reduce;
  };

  struct UninitializedSplit : hipcub::Uninitialized<Split> {};
  struct UninitializedGpair : hipcub::Uninitialized<gpu_gpair> {};

  __shared__ UninitializedSplit uninitialized_split;
  Split& split = uninitialized_split.Alias();
  __shared__ ArgMaxT block_max;
  __shared__ TempStorage temp_storage;

  if (threadIdx.x == 0) {
    split = Split();
  }

  __syncthreads();

  int node_idx = n_nodes(depth - 1) + blockIdx.x;

  for (int fidx = 0; fidx < n_features; fidx++) {
    if (colsample && d_feature_flags[fidx] == 0) continue;

    int begin = d_feature_segments[blockIdx.x * n_features + fidx];
    int end = d_feature_segments[blockIdx.x * n_features + fidx + 1];
    int gidx = (begin - (blockIdx.x * n_bins)) + threadIdx.x;
    bool thread_active = threadIdx.x < end - begin;

    // Scan histogram
    gpu_gpair bin =
        thread_active ? d_level_hist[begin + threadIdx.x] : gpu_gpair();

    gpu_gpair feature_sum;
    BlockScanT(temp_storage.scan)
        .ExclusiveScan(bin, bin, gpu_gpair(), hipcub::Sum(), feature_sum);

    // Calculate gain
    gpu_gpair parent_sum = d_nodes[node_idx].sum_gradients;
    float parent_gain = d_nodes[node_idx].root_gain;

    gpu_gpair missing = parent_sum - feature_sum;

    bool missing_left;
    float gain = thread_active
                     ? loss_chg_missing(bin, missing, parent_sum, parent_gain,
                                        gpu_param, missing_left)
                     : -FLT_MAX;
    __syncthreads();

    // Find thread with best gain
    ArgMaxT tuple(threadIdx.x, gain);
    ArgMaxT best = MaxReduceT(temp_storage.max_reduce)
                       .Reduce(tuple, hipcub::ArgMax(), end - begin);

    if (threadIdx.x == 0) {
      block_max = best;
    }

    __syncthreads();

    // Best thread updates split
    if (threadIdx.x == block_max.key) {
      float fvalue;
      if (threadIdx.x == 0) {
        fvalue = d_fidx_min_map[fidx];
      } else {
        fvalue = d_gidx_fvalue_map[gidx - 1];
      }

      gpu_gpair left = missing_left ? bin + missing : bin;
      gpu_gpair right = parent_sum - left;

      split.Update(gain, missing_left, fvalue, fidx, left, right, gpu_param);
    }
    __syncthreads();
  }

  // Create node
  if (threadIdx.x == 0) {
    d_nodes[node_idx].split = split;
    if (depth == 0) {
      // split.Print();
    }

    d_nodes[left_child_nidx(node_idx)] = Node(
        split.left_sum,
        CalcGain(gpu_param, split.left_sum.grad(), split.left_sum.hess()),
        CalcWeight(gpu_param, split.left_sum.grad(), split.left_sum.hess()));

    d_nodes[right_child_nidx(node_idx)] = Node(
        split.right_sum,
        CalcGain(gpu_param, split.right_sum.grad(), split.right_sum.hess()),
        CalcWeight(gpu_param, split.right_sum.grad(), split.right_sum.hess()));

    // Record smallest node
    if (split.left_sum.hess() <= split.right_sum.hess()) {
      d_left_child_smallest[node_idx] = true;
    } else {
      d_left_child_smallest[node_idx] = false;
    }
  }
}

void GPUHistBuilder::FindSplit(int depth) {
  // Specialised based on max_bins
  if (param.max_bin <= 256) {
    this->FindSplit256(depth);
  } else if (param.max_bin <= 1024) {
    this->FindSplit1024(depth);
  } else {
    this->FindSplitLarge(depth);
  }
}

void GPUHistBuilder::FindSplit256(int depth) {
  CHECK_LE(param.max_bin, 256);
  const int BLOCK_THREADS = 256;
  const int GRID_SIZE = n_nodes_level(depth);
  bool colsample =
      param.colsample_bylevel < 1.0 || param.colsample_bytree < 1.0;
  find_split_kernel<BLOCK_THREADS><<<GRID_SIZE, BLOCK_THREADS>>>(
      hist.GetLevelPtr(depth), feature_segments.data(), depth, info->num_col,
      hmat_.row_ptr.back(), nodes.data(), fidx_min_map.data(),
      gidx_fvalue_map.data(), gpu_param, left_child_smallest.data(), colsample,
      feature_flags.data());

  dh::safe_cuda(hipDeviceSynchronize());
}
void GPUHistBuilder::FindSplit1024(int depth) {
  CHECK_LE(param.max_bin, 1024);
  const int BLOCK_THREADS = 1024;
  const int GRID_SIZE = n_nodes_level(depth);
  bool colsample =
      param.colsample_bylevel < 1.0 || param.colsample_bytree < 1.0;
  find_split_kernel<BLOCK_THREADS><<<GRID_SIZE, BLOCK_THREADS>>>(
      hist.GetLevelPtr(depth), feature_segments.data(), depth, info->num_col,
      hmat_.row_ptr.back(), nodes.data(), fidx_min_map.data(),
      gidx_fvalue_map.data(), gpu_param, left_child_smallest.data(), colsample,
      feature_flags.data());

  dh::safe_cuda(hipDeviceSynchronize());
}
void GPUHistBuilder::FindSplitLarge(int depth) {
  auto counting = thrust::make_counting_iterator(0);
  auto d_gidx_feature_map = gidx_feature_map.data();
  int n_bins = hmat_.row_ptr.back();
  int n_features = hmat_.row_ptr.size() - 1;

  auto feature_boundary = [=] __device__(int idx_a, int idx_b) {
    int gidx_a = idx_a % n_bins;
    int gidx_b = idx_b % n_bins;
    return d_gidx_feature_map[gidx_a] == d_gidx_feature_map[gidx_b];
  };  // NOLINT

  // Reduce node sums
  {
    size_t temp_storage_bytes;
    hipcub::DeviceSegmentedReduce::Reduce(
        nullptr, temp_storage_bytes, hist.GetLevelPtr(depth), node_sums.data(),
        n_nodes_level(depth) * n_features, feature_segments.data(),
        feature_segments.data() + 1, hipcub::Sum(), gpu_gpair());
    cub_mem.LazyAllocate(temp_storage_bytes);
    hipcub::DeviceSegmentedReduce::Reduce(
        cub_mem.d_temp_storage, cub_mem.temp_storage_bytes,
        hist.GetLevelPtr(depth), node_sums.data(),
        n_nodes_level(depth) * n_features, feature_segments.data(),
        feature_segments.data() + 1, hipcub::Sum(), gpu_gpair());
  }

  // Scan
  thrust::exclusive_scan_by_key(
      counting, counting + hist.LevelSize(depth),
      thrust::device_pointer_cast(hist.GetLevelPtr(depth)), hist_scan.tbegin(),
      gpu_gpair(), feature_boundary);

  // Calculate gain
  auto d_gain = gain.data();
  auto d_nodes = nodes.data();
  auto d_node_sums = node_sums.data();
  auto d_hist_scan = hist_scan.data();
  GPUTrainingParam gpu_param_alias =
      gpu_param;  // Must be local variable to be used in device lambda
  bool colsample =
      param.colsample_bylevel < 1.0 || param.colsample_bytree < 1.0;
  auto d_feature_flags = feature_flags.data();

  dh::launch_n(hist.LevelSize(depth), [=] __device__(int idx) {
    int node_segment = idx / n_bins;
    int node_idx = n_nodes(depth - 1) + node_segment;
    gpu_gpair parent_sum = d_nodes[node_idx].sum_gradients;
    float parent_gain = d_nodes[node_idx].root_gain;
    int gidx = idx % n_bins;
    int findex = d_gidx_feature_map[gidx];

    // colsample
    if (colsample && d_feature_flags[d_gidx_feature_map[gidx]] == 0) {
      d_gain[idx] = 0;
    } else {
      gpu_gpair scan = d_hist_scan[idx];
      gpu_gpair sum = d_node_sums[node_segment * n_features + findex];
      gpu_gpair missing = parent_sum - sum;

      bool missing_left;
      d_gain[idx] = loss_chg_missing(scan, missing, parent_sum, parent_gain,
                                     gpu_param_alias, missing_left);
    }
  });
  dh::safe_cuda(hipDeviceSynchronize());

  // Find best gain
  {
    size_t temp_storage_bytes;
    hipcub::DeviceSegmentedReduce::ArgMax(nullptr, temp_storage_bytes, gain.data(),
                                       argmax.data(), n_nodes_level(depth),
                                       hist_node_segments.data(),
                                       hist_node_segments.data() + 1);
    cub_mem.LazyAllocate(temp_storage_bytes);
    hipcub::DeviceSegmentedReduce::ArgMax(
        cub_mem.d_temp_storage, cub_mem.temp_storage_bytes, gain.data(),
        argmax.data(), n_nodes_level(depth), hist_node_segments.data(),
        hist_node_segments.data() + 1);
  }

  auto d_argmax = argmax.data();
  auto d_gidx_fvalue_map = gidx_fvalue_map.data();
  auto d_fidx_min_map = fidx_min_map.data();
  auto d_left_child_smallest = left_child_smallest.data();

  dh::launch_n(n_nodes_level(depth), [=] __device__(int idx) {
    int max_idx = n_bins * idx + d_argmax[idx].key;
    int gidx = max_idx % n_bins;
    int fidx = d_gidx_feature_map[gidx];
    int node_segment = max_idx / n_bins;
    int node_idx = n_nodes(depth - 1) + node_segment;
    gpu_gpair scan = d_hist_scan[max_idx];
    gpu_gpair parent_sum = d_nodes[node_idx].sum_gradients;
    float parent_gain = d_nodes[node_idx].root_gain;
    gpu_gpair sum = d_node_sums[node_segment * n_features + fidx];
    gpu_gpair missing = parent_sum - sum;

    bool missing_left;
    float loss_chg = loss_chg_missing(scan, missing, parent_sum, parent_gain,
                                      gpu_param_alias, missing_left);

    float fvalue;
    if (gidx == 0 || fidx != d_gidx_feature_map[gidx - 1]) {
      fvalue = d_fidx_min_map[fidx];
    } else {
      fvalue = d_gidx_fvalue_map[gidx - 1];
    }
    gpu_gpair left = missing_left ? scan + missing : scan;
    gpu_gpair right = parent_sum - left;
    d_nodes[node_idx].split.Update(loss_chg, missing_left, fvalue, fidx, left,
                                   right, gpu_param_alias);

    d_nodes[left_child_nidx(node_idx)] =
        Node(left, CalcGain(gpu_param_alias, left.grad(), left.hess()),
             CalcWeight(gpu_param_alias, left.grad(), left.hess()));

    d_nodes[right_child_nidx(node_idx)] =
        Node(right, CalcGain(gpu_param_alias, right.grad(), right.hess()),
             CalcWeight(gpu_param_alias, right.grad(), right.hess()));

    // Record smallest node
    if (left.hess() <= right.hess()) {
      d_left_child_smallest[node_idx] = true;
    } else {
      d_left_child_smallest[node_idx] = false;
    }
  });
  dh::safe_cuda(hipDeviceSynchronize());
}

void GPUHistBuilder::InitFirstNode() {
  auto d_gpair = device_gpair.data();
  auto d_node_sums = node_sums.data();
  auto d_nodes = nodes.data();
  auto gpu_param_alias = gpu_param;

  size_t temp_storage_bytes;
  hipcub::DeviceReduce::Reduce(nullptr, temp_storage_bytes, d_gpair, d_node_sums,
                            device_gpair.size(), hipcub::Sum(), gpu_gpair());
  cub_mem.LazyAllocate(temp_storage_bytes);
  hipcub::DeviceReduce::Reduce(cub_mem.d_temp_storage, cub_mem.temp_storage_bytes,
                            d_gpair, d_node_sums, device_gpair.size(),
                            hipcub::Sum(), gpu_gpair());

  dh::launch_n(1, [=] __device__(int idx) {
    gpu_gpair sum_gradients = d_node_sums[idx];
    d_nodes[idx] = Node(
        sum_gradients,
        CalcGain(gpu_param_alias, sum_gradients.grad(), sum_gradients.hess()),
        CalcWeight(gpu_param_alias, sum_gradients.grad(),
                   sum_gradients.hess()));
  });
}

void GPUHistBuilder::UpdatePosition(int depth) {
  if (is_dense) {
    this->UpdatePositionDense(depth);
  } else {
    this->UpdatePositionSparse(depth);
  }
}

void GPUHistBuilder::UpdatePositionDense(int depth) {
  auto d_position = position.data();
  Node* d_nodes = nodes.data();
  auto d_gidx_fvalue_map = gidx_fvalue_map.data();
  auto d_gidx = device_matrix.gidx.data();
  int n_columns = info->num_col;

  int gidx_size = device_matrix.gidx.size();

  dh::launch_n(position.size(), [=] __device__(int idx) {
    NodeIdT pos = d_position[idx];
    if (!is_active(pos, depth)) {
      return;
    }
    Node node = d_nodes[pos];

    if (node.IsLeaf()) {
      return;
    }

    int gidx = d_gidx[idx * n_columns + node.split.findex];

    float fvalue = d_gidx_fvalue_map[gidx];

    if (fvalue <= node.split.fvalue) {
      d_position[idx] = left_child_nidx(pos);
    } else {
      d_position[idx] = right_child_nidx(pos);
    }
  });

  dh::safe_cuda(hipDeviceSynchronize());
}

void GPUHistBuilder::UpdatePositionSparse(int depth) {
  auto d_position = position.data();
  auto d_position_tmp = position_tmp.data();
  Node* d_nodes = nodes.data();
  auto d_gidx_feature_map = gidx_feature_map.data();
  auto d_gidx_fvalue_map = gidx_fvalue_map.data();
  auto d_gidx = device_matrix.gidx.data();
  auto d_ridx = device_matrix.ridx.data();

  // Update missing direction
  dh::launch_n(position.size(), [=] __device__(int idx) {
    NodeIdT pos = d_position[idx];
    if (!is_active(pos, depth)) {
      d_position_tmp[idx] = pos;
      return;
    }

    Node node = d_nodes[pos];

    if (node.IsLeaf()) {
      d_position_tmp[idx] = pos;
      return;
    } else if (node.split.missing_left) {
      d_position_tmp[idx] = pos * 2 + 1;
    } else {
      d_position_tmp[idx] = pos * 2 + 2;
    }
  });

  dh::safe_cuda(hipDeviceSynchronize());

  // Update node based on fvalue where exists
  dh::launch_n(device_matrix.gidx.size(), [=] __device__(int idx) {
    int ridx = d_ridx[idx];
    NodeIdT pos = d_position[ridx];
    if (!is_active(pos, depth)) {
      return;
    }

    Node node = d_nodes[pos];

    if (node.IsLeaf()) {
      return;
    }

    int gidx = d_gidx[idx];
    int findex = d_gidx_feature_map[gidx];

    if (findex == node.split.findex) {
      float fvalue = d_gidx_fvalue_map[gidx];

      if (fvalue <= node.split.fvalue) {
        d_position_tmp[ridx] = left_child_nidx(pos);
      } else {
        d_position_tmp[ridx] = right_child_nidx(pos);
      }
    }
  });

  dh::safe_cuda(hipDeviceSynchronize());

  position = position_tmp;
}

void GPUHistBuilder::ColSampleTree() {
  if (param.colsample_bylevel == 1.0 && param.colsample_bytree == 1.0) return;

  feature_set_tree.resize(info->num_col);
  std::iota(feature_set_tree.begin(), feature_set_tree.end(), 0);
  feature_set_tree = col_sample(feature_set_tree, param.colsample_bytree);
}

void GPUHistBuilder::ColSampleLevel() {
  if (param.colsample_bylevel == 1.0 && param.colsample_bytree == 1.0) return;

  feature_set_level.resize(feature_set_tree.size());
  feature_set_level = col_sample(feature_set_tree, param.colsample_bylevel);
  std::vector<int> h_feature_flags(info->num_col, 0);
  for (auto fidx : feature_set_level) {
    h_feature_flags[fidx] = 1;
  }
  feature_flags = h_feature_flags;
}

void GPUHistBuilder::InitData(const std::vector<bst_gpair>& gpair,
                              DMatrix& fmat,  // NOLINT
                              const RegTree& tree) {
  if (!initialised) {
    CHECK(fmat.SingleColBlock()) << "grow_gpu_hist: must have single column "
                                    "block. Try setting 'tree_method' "
                                    "parameter to 'exact'";
    info = &fmat.info();
    is_dense = info->num_nonzero == info->num_col * info->num_row;
    hmat_.Init(&fmat, param.max_bin);
    gmat_.cut = &hmat_;
    gmat_.Init(&fmat);
    int n_bins = hmat_.row_ptr.back();
    int n_features = hmat_.row_ptr.size() - 1;

    // Build feature segments
    std::vector<int> h_feature_segments;
    for (int node = 0; node < n_nodes_level(param.max_depth - 1); node++) {
      for (int fidx = 0; fidx < hmat_.row_ptr.size() - 1; fidx++) {
        h_feature_segments.push_back(hmat_.row_ptr[fidx] + node * n_bins);
      }
    }
    h_feature_segments.push_back(n_nodes_level(param.max_depth - 1) * n_bins);

    int level_max_bins = n_nodes_level(param.max_depth - 1) * n_bins;

    size_t free_memory = dh::available_memory();
    ba.allocate(&gidx_feature_map, n_bins, &hist_node_segments,
                n_nodes_level(param.max_depth - 1) + 1, &feature_segments,
                h_feature_segments.size(), &gain, level_max_bins, &position,
                gpair.size(), &position_tmp, gpair.size(), &nodes,
                n_nodes(param.max_depth), &gidx_fvalue_map, hmat_.cut.size(),
                &fidx_min_map, hmat_.min_val.size(), &argmax,
                n_nodes_level(param.max_depth - 1), &node_sums,
                n_nodes_level(param.max_depth - 1) * n_features, &hist_scan,
                level_max_bins, &device_gpair, gpair.size(),
                &device_matrix.gidx, gmat_.index.size(), &device_matrix.ridx,
                gmat_.index.size(), &hist.hist,
                n_nodes(param.max_depth - 1) * n_bins, &feature_flags,
                n_features, &left_child_smallest, n_nodes(param.max_depth - 1),
                &prediction_cache, gpair.size());

    if (!param.silent) {
      const int mb_size = 1048576;
      LOG(CONSOLE) << "Allocated " << ba.size() / mb_size << "/"
                   << free_memory / mb_size << " MB on " << dh::device_name();
    }

    // Construct feature map
    std::vector<int> h_gidx_feature_map(n_bins);
    for (int row = 0; row < hmat_.row_ptr.size() - 1; row++) {
      for (int i = hmat_.row_ptr[row]; i < hmat_.row_ptr[row + 1]; i++) {
        h_gidx_feature_map[i] = row;
      }
    }

    gidx_feature_map = h_gidx_feature_map;

    // Construct device matrix
    device_matrix.Init(gmat_);

    gidx_fvalue_map = hmat_.cut;
    fidx_min_map = hmat_.min_val;

    thrust::sequence(hist_node_segments.tbegin(), hist_node_segments.tend(), 0,
                     n_bins);

    feature_flags.fill(1);

    feature_segments = h_feature_segments;

    hist.Init(n_bins);

    prediction_cache.fill(0);

    initialised = true;
  }
  nodes.fill(Node());
  position.fill(0);
  device_gpair = gpair;
  subsample_gpair(&device_gpair, param.subsample);
  hist.Reset();
  p_last_fmat_ = &fmat;
}

bool GPUHistBuilder::UpdatePredictionCache(
    const DMatrix* data, std::vector<bst_float>* p_out_preds) {
  std::vector<bst_float>& out_preds = *p_out_preds;

  if (nodes.empty() || !p_last_fmat_ || data != p_last_fmat_) {
    return false;
  }
  CHECK_EQ(prediction_cache.size(), out_preds.size());

  if (!prediction_cache_initialised) {
    prediction_cache = out_preds;
    prediction_cache_initialised = true;
  }

  auto d_nodes = nodes.data();
  auto d_position = position.data();
  auto d_prediction_cache = prediction_cache.data();
  float eps = param.learning_rate;

  dh::launch_n(prediction_cache.size(), [=] __device__(int idx) {
    int pos = d_position[idx];
    d_prediction_cache[idx] += d_nodes[pos].weight * eps;
  });

  thrust::copy(prediction_cache.tbegin(), prediction_cache.tend(),
               out_preds.data());

  return true;
}

void GPUHistBuilder::Update(const std::vector<bst_gpair>& gpair,
                            DMatrix* p_fmat, RegTree* p_tree) {
  this->InitData(gpair, *p_fmat, *p_tree);
  this->InitFirstNode();
  this->ColSampleTree();
  for (int depth = 0; depth < param.max_depth; depth++) {
    this->ColSampleLevel();
    this->BuildHist(depth);
    this->FindSplit(depth);
    this->UpdatePosition(depth);
  }
  dense2sparse_tree(p_tree, nodes.tbegin(), nodes.tend(), param);
}
}  // namespace tree
}  // namespace xgboost
