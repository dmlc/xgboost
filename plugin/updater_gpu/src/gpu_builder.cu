/*!
 * Copyright 2016 Rory mitchell
*/
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <algorithm>
#include <random>
#include <numeric>
#include <vector>
#include "../../../src/common/random.h"
#include "device_helpers.cuh"
#include "find_split.cuh"
#include "gpu_builder.cuh"
#include "types_functions.cuh"
#include "gpu_data.cuh"

namespace xgboost {
namespace tree {

GPUBuilder::GPUBuilder() { gpu_data = new GPUData(); }

void GPUBuilder::Init(const TrainParam &param_in) {
  param = param_in;
  CHECK(param.max_depth < 16) << "Tree depth too large.";
}

GPUBuilder::~GPUBuilder() { delete gpu_data; }

void GPUBuilder::UpdateNodeId(int level) {
  auto *d_node_id_instance = gpu_data->node_id_instance.data();
  Node *d_nodes = gpu_data->nodes.data();

  dh::launch_n(gpu_data->node_id_instance.size(), [=] __device__(int i) {
    NodeIdT item_node_id = d_node_id_instance[i];

    if (item_node_id < 0) {
      return;
    }

    Node node = d_nodes[item_node_id];

    if (node.IsLeaf()) {
      d_node_id_instance[i] = -1;
    } else if (node.split.missing_left) {
      d_node_id_instance[i] = item_node_id * 2 + 1;
    } else {
      d_node_id_instance[i] = item_node_id * 2 + 2;
    }
  });

  dh::safe_cuda(hipDeviceSynchronize());

  auto *d_fvalues = gpu_data->fvalues.data();
  auto *d_instance_id = gpu_data->instance_id.data();
  auto *d_node_id = gpu_data->node_id.data();
  auto *d_feature_id = gpu_data->feature_id.data();

  // Update node based on fvalue where exists
  dh::launch_n(gpu_data->fvalues.size(), [=] __device__(int i) {
    NodeIdT item_node_id = d_node_id[i];

    if (item_node_id < 0) {
      return;
    }

    Node node = d_nodes[item_node_id];

    if (node.IsLeaf()) {
      return;
    }

    int feature_id = d_feature_id[i];

    if (feature_id == node.split.findex) {
      float fvalue = d_fvalues[i];
      bst_uint instance_id = d_instance_id[i];

      if (fvalue < node.split.fvalue) {
        d_node_id_instance[instance_id] = item_node_id * 2 + 1;
      } else {
        d_node_id_instance[instance_id] = item_node_id * 2 + 2;
      }
    }
  });

  dh::safe_cuda(hipDeviceSynchronize());

  gpu_data->GatherNodeId();
}

void GPUBuilder::Sort(int level) {
  thrust::sequence(gpu_data->sort_index_in.tbegin(),
                   gpu_data->sort_index_in.tend());

  hipcub::DoubleBuffer<NodeIdT> d_keys(gpu_data->node_id.data(),
                                    gpu_data->node_id_temp.data());
  hipcub::DoubleBuffer<int> d_values(gpu_data->sort_index_in.data(),
                                  gpu_data->sort_index_out.data());

  size_t temp_size = gpu_data->cub_mem.size();

  hipcub::DeviceSegmentedRadixSort::SortPairs(
      gpu_data->cub_mem.data(), temp_size, d_keys, d_values,
      gpu_data->fvalues.size(), gpu_data->n_features, gpu_data->foffsets.data(),
      gpu_data->foffsets.data() + 1);

  auto zip = thrust::make_zip_iterator(thrust::make_tuple(
      gpu_data->fvalues.tbegin(), gpu_data->instance_id.tbegin()));
  auto zip_temp = thrust::make_zip_iterator(thrust::make_tuple(
      gpu_data->fvalues_temp.tbegin(), gpu_data->instance_id_temp.tbegin()));
  thrust::gather(thrust::device_pointer_cast(d_values.Current()),
                 thrust::device_pointer_cast(d_values.Current()) +
                     gpu_data->sort_index_out.size(),
                 zip, zip_temp);
  thrust::copy(zip_temp, zip_temp + gpu_data->fvalues.size(), zip);

  if (d_keys.Current() == gpu_data->node_id_temp.data()) {
    thrust::copy(gpu_data->node_id_temp.tbegin(), gpu_data->node_id_temp.tend(),
                 gpu_data->node_id.tbegin());
  }
}

void GPUBuilder::ColsampleTree() {
  unsigned n = static_cast<unsigned>(
    param.colsample_bytree * gpu_data->n_features);
  CHECK_GT(n, 0);

  feature_set_tree.resize(gpu_data->n_features);
  std::iota(feature_set_tree.begin(), feature_set_tree.end(), 0);
  std::shuffle(feature_set_tree.begin(), feature_set_tree.end(),
    common::GlobalRandom());
}

void GPUBuilder::Update(const std::vector<bst_gpair> &gpair, DMatrix *p_fmat,
                        RegTree *p_tree) {
  try {
    dh::Timer update;
    dh::Timer t;
    this->InitData(gpair, *p_fmat, *p_tree);
    t.printElapsed("init data");
    this->InitFirstNode();
    this->ColsampleTree();

    for (int level = 0; level < param.max_depth; level++) {
      bool use_multiscan_algorithm = level < multiscan_levels;

      t.reset();
      if (level > 0) {
        dh::Timer update_node;
        this->UpdateNodeId(level);
        update_node.printElapsed("node");
      }

      if (level > 0 && !use_multiscan_algorithm) {
        dh::Timer s;
        this->Sort(level);
        s.printElapsed("sort");
      }

      dh::Timer split;
      find_split(gpu_data, param, level, use_multiscan_algorithm,
        feature_set_tree, &feature_set_level);

      split.printElapsed("split");

      t.printElapsed("level");
    }
    this->CopyTree(*p_tree);
    update.printElapsed("update");
  } catch (thrust::system_error &e) {
    std::cerr << "CUDA error: " << e.what() << std::endl;
    exit(-1);
  } catch (const std::exception &e) {
    std::cerr << "Error: " << e.what() << std::endl;
    exit(-1);
  } catch (...) {
    std::cerr << "Unknown exception." << std::endl;
    exit(-1);
  }
}

void GPUBuilder::InitData(const std::vector<bst_gpair> &gpair, DMatrix &fmat,
                          const RegTree &tree) {
  CHECK(fmat.SingleColBlock()) << "GPUMaker: must have single column block";

  if (gpu_data->IsAllocated()) {
    gpu_data->Reset(gpair, param.subsample);
    return;
  }

  dh::Timer t;

  MetaInfo info = fmat.info();

  std::vector<int> foffsets;
  foffsets.push_back(0);
  std::vector<int> feature_id;
  std::vector<float> fvalues;
  std::vector<bst_uint> instance_id;
  fvalues.reserve(info.num_col * info.num_row);
  instance_id.reserve(info.num_col * info.num_row);
  feature_id.reserve(info.num_col * info.num_row);

  dmlc::DataIter<ColBatch> *iter = fmat.ColIterator();

  while (iter->Next()) {
    const ColBatch &batch = iter->Value();

    for (int i = 0; i < batch.size; i++) {
      const ColBatch::Inst &col = batch[i];

      for (const ColBatch::Entry *it = col.data; it != col.data + col.length;
           it++) {
        bst_uint inst_id = it->index;
          fvalues.push_back(it->fvalue);
          instance_id.push_back(inst_id);
          feature_id.push_back(i);
      }
      foffsets.push_back(fvalues.size());
    }
  }

  t.printElapsed("dmatrix");
  t.reset();
  gpu_data->Init(fvalues, foffsets, instance_id, feature_id, gpair,
                 info.num_row, info.num_col, param.max_depth, param);

  t.printElapsed("gpu init");
}

void GPUBuilder::InitFirstNode() {
  // Build the root node on the CPU and copy to device
  gpu_gpair sum_gradients =
      thrust::reduce(gpu_data->gpair.tbegin(), gpu_data->gpair.tend(),
                     gpu_gpair(0, 0), hipcub::Sum());

  Node tmp = Node(
      sum_gradients,
      CalcGain(gpu_data->param, sum_gradients.grad(), sum_gradients.hess()),
      CalcWeight(gpu_data->param, sum_gradients.grad(), sum_gradients.hess()));

  thrust::copy_n(&tmp, 1, gpu_data->nodes.tbegin());
}

enum NodeType {
  NODE = 0,
  LEAF = 1,
  UNUSED = 2,
};

// Recursively label node types
void flag_nodes(const thrust::host_vector<Node> &nodes,
                std::vector<NodeType> *node_flags, int nid, NodeType type) {
  if (nid >= nodes.size() || type == UNUSED) {
    return;
  }

  const Node &n = nodes[nid];

  // Current node and all children are valid
  if (n.split.loss_chg > rt_eps) {
    (*node_flags)[nid] = NODE;
    flag_nodes(nodes, node_flags, nid * 2 + 1, NODE);
    flag_nodes(nodes, node_flags, nid * 2 + 2, NODE);
  } else {
    // Current node is leaf, therefore is valid but all children are invalid
    (*node_flags)[nid] = LEAF;
    flag_nodes(nodes, node_flags, nid * 2 + 1, UNUSED);
    flag_nodes(nodes, node_flags, nid * 2 + 2, UNUSED);
  }
}

// Copy gpu dense representation of tree to xgboost sparse representation
void GPUBuilder::CopyTree(RegTree &tree) {
  std::vector<Node> h_nodes = gpu_data->nodes.as_vector();
  std::vector<NodeType> node_flags(h_nodes.size(), UNUSED);
  flag_nodes(h_nodes, &node_flags, 0, NODE);

  int nid = 0;
  for (int gpu_nid = 0; gpu_nid < h_nodes.size(); gpu_nid++) {
    NodeType flag = node_flags[gpu_nid];
    const Node &n = h_nodes[gpu_nid];
    if (flag == NODE) {
      tree.AddChilds(nid);
      tree[nid].set_split(n.split.findex, n.split.fvalue, n.split.missing_left);
      tree.stat(nid).loss_chg = n.split.loss_chg;
      tree.stat(nid).base_weight = n.weight;
      tree.stat(nid).sum_hess = n.sum_gradients.hess();
      tree[tree[nid].cleft()].set_leaf(0);
      tree[tree[nid].cright()].set_leaf(0);
      nid++;
    } else if (flag == LEAF) {
      tree[nid].set_leaf(n.weight * param.learning_rate);
      tree.stat(nid).sum_hess = n.sum_gradients.hess();
      nid++;
    }
  }
}
}  // namespace tree
}  // namespace xgboost
