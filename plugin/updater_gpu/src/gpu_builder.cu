#include "hip/hip_runtime.h"
/*!
 * Copyright 2016 Rory mitchell
*/
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <algorithm>
#include <random>
#include <vector>
#include "../../../src/common/random.h"
#include "device_helpers.cuh"
#include "find_split.cuh"
#include "gpu_builder.cuh"
#include "types_functions.cuh"

namespace xgboost {
namespace tree {
struct GPUData {
  GPUData() : allocated(false), n_features(0), n_instances(0) {}

  bool allocated;
  int n_features;
  int n_instances;

  dh::bulk_allocator ba;
  GPUTrainingParam param;

  dh::dvec<float> fvalues;
  dh::dvec<float> fvalues_temp;
  dh::dvec<float> fvalues_cached;
  dh::dvec<int> foffsets;
  dh::dvec<bst_uint> instance_id;
  dh::dvec<bst_uint> instance_id_temp;
  dh::dvec<bst_uint> instance_id_cached;
  dh::dvec<int> feature_id;
  dh::dvec<NodeIdT> node_id;
  dh::dvec<NodeIdT> node_id_temp;
  dh::dvec<NodeIdT> node_id_instance;
  dh::dvec<gpu_gpair> gpair;
  dh::dvec<Node> nodes;
  dh::dvec<Split> split_candidates;
  dh::dvec<gpu_gpair> node_sums;
  dh::dvec<int> node_offsets;
  dh::dvec<int> sort_index_in;
  dh::dvec<int> sort_index_out;

  dh::dvec<char> cub_mem;

  ItemIter items_iter;

  void Init(const std::vector<float> &in_fvalues,
            const std::vector<int> &in_foffsets,
            const std::vector<bst_uint> &in_instance_id,
            const std::vector<int> &in_feature_id,
            const std::vector<bst_gpair> &in_gpair, bst_uint n_instances_in,
            bst_uint n_features_in, int max_depth, const TrainParam &param_in) {
    n_features = n_features_in;
    n_instances = n_instances_in;

    uint32_t max_nodes = (1 << (max_depth + 1)) - 1;
    uint32_t max_nodes_level = 1 << max_depth;

    // Calculate memory for sort
    size_t cub_mem_size = 0;
    hipcub::DeviceSegmentedRadixSort::SortPairs(
        cub_mem.data(), cub_mem_size, hipcub::DoubleBuffer<NodeIdT>(),
        hipcub::DoubleBuffer<int>(), in_fvalues.size(), n_features,
        foffsets.data(), foffsets.data() + 1);

    // Allocate memory
    size_t free_memory = dh::available_memory();
    ba.allocate(&fvalues, in_fvalues.size(), &fvalues_temp, in_fvalues.size(),
                &fvalues_cached, in_fvalues.size(), &foffsets,
                in_foffsets.size(), &instance_id, in_instance_id.size(),
                &instance_id_temp, in_instance_id.size(), &instance_id_cached,
                in_instance_id.size(), &feature_id, in_feature_id.size(),
                &node_id, in_fvalues.size(), &node_id_temp, in_fvalues.size(),
                &node_id_instance, n_instances, &gpair, n_instances, &nodes,
                max_nodes, &split_candidates, max_nodes_level * n_features,
                &node_sums, max_nodes_level * n_features, &node_offsets,
                max_nodes_level * n_features, &sort_index_in, in_fvalues.size(),
                &sort_index_out, in_fvalues.size(), &cub_mem, cub_mem_size);

    if (!param_in.silent) {
      const int mb_size = 1048576;
      LOG(CONSOLE) << "Allocated " << ba.size() / mb_size << "/"
                   << free_memory / mb_size << " MB on " << dh::device_name();
    }
    node_id.fill(0);
    node_id_instance.fill(0);

    fvalues = in_fvalues;
    fvalues_cached = fvalues;
    foffsets = in_foffsets;
    instance_id = in_instance_id;
    instance_id_cached = instance_id;
    feature_id = in_feature_id;

    param = GPUTrainingParam(param_in.min_child_weight, param_in.reg_lambda,
                             param_in.reg_alpha, param_in.max_delta_step);

    gpair = in_gpair;

    nodes.fill(Node());

    items_iter = thrust::make_zip_iterator(thrust::make_tuple(
        thrust::make_permutation_iterator(gpair.tbegin(), instance_id.tbegin()),
        fvalues.tbegin(), node_id.tbegin()));

    allocated = true;

    dh::safe_cuda(hipGetLastError());
  }

  ~GPUData() {}

  // Reset memory for new boosting iteration
  void Reset(const std::vector<bst_gpair> &in_gpair) {
    CHECK(allocated);
    gpair = in_gpair;
    instance_id = instance_id_cached;
    fvalues = fvalues_cached;
    nodes.fill(Node());
    node_id_instance.fill(0);
    node_id.fill(0);
  }

  bool IsAllocated() { return allocated; }

  // Gather from node_id_instance into node_id according to instance_id
  void GatherNodeId() {
    // Update node_id for each item
    auto d_node_id = node_id.data();
    auto d_node_id_instance = node_id_instance.data();
    auto d_instance_id = instance_id.data();

    dh::launch_n(fvalues.size(), [=] __device__(bst_uint i) {
      // Item item = d_items[i];
      d_node_id[i] = d_node_id_instance[d_instance_id[i]];
    });
  }
};

GPUBuilder::GPUBuilder() { gpu_data = new GPUData(); }

void GPUBuilder::Init(const TrainParam &param_in) {
  param = param_in;
  CHECK(param.max_depth < 16) << "Tree depth too large.";
}

GPUBuilder::~GPUBuilder() { delete gpu_data; }

void GPUBuilder::UpdateNodeId(int level) {
  auto *d_node_id_instance = gpu_data->node_id_instance.data();
  Node *d_nodes = gpu_data->nodes.data();

  dh::launch_n(gpu_data->node_id_instance.size(), [=] __device__(int i) {
    NodeIdT item_node_id = d_node_id_instance[i];

    if (item_node_id < 0) {
      return;
    }

    Node node = d_nodes[item_node_id];

    if (node.IsLeaf()) {
      d_node_id_instance[i] = -1;
    } else if (node.split.missing_left) {
      d_node_id_instance[i] = item_node_id * 2 + 1;
    } else {
      d_node_id_instance[i] = item_node_id * 2 + 2;
    }
  });

  dh::safe_cuda(hipDeviceSynchronize());

  auto *d_fvalues = gpu_data->fvalues.data();
  auto *d_instance_id = gpu_data->instance_id.data();
  auto *d_node_id = gpu_data->node_id.data();
  auto *d_feature_id = gpu_data->feature_id.data();

  // Update node based on fvalue where exists
  dh::launch_n(gpu_data->fvalues.size(), [=] __device__(int i) {
    NodeIdT item_node_id = d_node_id[i];

    if (item_node_id < 0) {
      return;
    }

    Node node = d_nodes[item_node_id];

    if (node.IsLeaf()) {
      return;
    }

    int feature_id = d_feature_id[i];

    if (feature_id == node.split.findex) {
      float fvalue = d_fvalues[i];
      bst_uint instance_id = d_instance_id[i];

      if (fvalue < node.split.fvalue) {
        d_node_id_instance[instance_id] = item_node_id * 2 + 1;
      } else {
        d_node_id_instance[instance_id] = item_node_id * 2 + 2;
      }
    }
  });

  dh::safe_cuda(hipDeviceSynchronize());

  gpu_data->GatherNodeId();
}

void GPUBuilder::Sort(int level) {
  thrust::sequence(gpu_data->sort_index_in.tbegin(),
                   gpu_data->sort_index_in.tend());

  hipcub::DoubleBuffer<NodeIdT> d_keys(gpu_data->node_id.data(),
                                    gpu_data->node_id_temp.data());
  hipcub::DoubleBuffer<int> d_values(gpu_data->sort_index_in.data(),
                                  gpu_data->sort_index_out.data());

  size_t temp_size = gpu_data->cub_mem.size();

  hipcub::DeviceSegmentedRadixSort::SortPairs(
      gpu_data->cub_mem.data(), temp_size, d_keys, d_values,
      gpu_data->fvalues.size(), gpu_data->n_features, gpu_data->foffsets.data(),
      gpu_data->foffsets.data() + 1);

  auto zip = thrust::make_zip_iterator(thrust::make_tuple(
      gpu_data->fvalues.tbegin(), gpu_data->instance_id.tbegin()));
  auto zip_temp = thrust::make_zip_iterator(thrust::make_tuple(
      gpu_data->fvalues_temp.tbegin(), gpu_data->instance_id_temp.tbegin()));
  thrust::gather(thrust::device_pointer_cast(d_values.Current()),
                 thrust::device_pointer_cast(d_values.Current()) +
                     gpu_data->sort_index_out.size(),
                 zip, zip_temp);
  thrust::copy(zip_temp, zip_temp + gpu_data->fvalues.size(), zip);

  if (d_keys.Current() == gpu_data->node_id_temp.data()) {
    thrust::copy(gpu_data->node_id_temp.tbegin(), gpu_data->node_id_temp.tend(),
                 gpu_data->node_id.tbegin());
  }
}

void GPUBuilder::Update(const std::vector<bst_gpair> &gpair, DMatrix *p_fmat,
                        RegTree *p_tree) {
  hipProfilerStart();
  try {
    dh::Timer update;
    dh::Timer t;
    this->InitData(gpair, *p_fmat, *p_tree);
    t.printElapsed("init data");
    this->InitFirstNode();

    for (int level = 0; level < param.max_depth; level++) {
      bool use_multiscan_algorithm = level < multiscan_levels;

      t.reset();
      if (level > 0) {
        dh::Timer update_node;
        this->UpdateNodeId(level);
        update_node.printElapsed("node");
      }

      if (level > 0 && !use_multiscan_algorithm) {
        dh::Timer s;
        this->Sort(level);
        s.printElapsed("sort");
      }

      dh::Timer split;
      find_split(gpu_data->items_iter, gpu_data->split_candidates.data(),
                 gpu_data->nodes.data(), (bst_uint)gpu_data->fvalues.size(),
                 gpu_data->n_features, gpu_data->foffsets.data(),
                 gpu_data->node_sums.data(), gpu_data->node_offsets.data(),
                 gpu_data->param, level, use_multiscan_algorithm);

      split.printElapsed("split");

      t.printElapsed("level");
    }
    this->CopyTree(*p_tree);
    update.printElapsed("update");
  } catch (thrust::system_error &e) {
    std::cerr << "CUDA error: " << e.what() << std::endl;
    exit(-1);
  } catch (const std::exception &e) {
    std::cerr << "Error: " << e.what() << std::endl;
    exit(-1);
  } catch (...) {
    std::cerr << "Unknown exception." << std::endl;
    exit(-1);
  }
  hipProfilerStop();
}

float GPUBuilder::GetSubsamplingRate(MetaInfo info) {
  float subsample = 1.0;
  size_t required = 10 * info.num_row + 44 * info.num_nonzero;
  size_t available = dh::available_memory();
  while (available < required) {
    subsample -= 0.05;
    required = 10 * info.num_row + subsample * (44 * info.num_nonzero);
  }

  return subsample;
}

void GPUBuilder::InitData(const std::vector<bst_gpair> &gpair, DMatrix &fmat,
                          const RegTree &tree) {
  CHECK(fmat.SingleColBlock()) << "GPUMaker: must have single column block";

  if (gpu_data->IsAllocated()) {
    gpu_data->Reset(gpair);
    return;
  }

  dh::Timer t;

  MetaInfo info = fmat.info();

  // Work out if dataset will fit on GPU
  float subsample = this->GetSubsamplingRate(info);
  CHECK(subsample > 0.0);
  if (!param.silent && subsample < param.subsample) {
    LOG(CONSOLE) << "Not enough device memory for entire dataset.";
  }

  // Override subsample parameter if user-specified parameter is lower
  subsample = std::min(param.subsample, subsample);

  std::vector<bool> row_flags;

  if (subsample < 1.0) {
    if (!param.silent && subsample < 1.0) {
      LOG(CONSOLE) << "Subsampling " << subsample * 100 << "% of rows.";
    }

    const RowSet &rowset = fmat.buffered_rowset();
    row_flags.resize(info.num_row);
    std::bernoulli_distribution coin_flip(subsample);
    auto &rnd = common::GlobalRandom();
    for (size_t i = 0; i < rowset.size(); ++i) {
      const bst_uint ridx = rowset[i];
      if (gpair[ridx].hess < 0.0f)
        continue;
      row_flags[ridx] = coin_flip(rnd);
    }
  }

  std::vector<int> foffsets;
  foffsets.push_back(0);
  std::vector<int> feature_id;
  std::vector<float> fvalues;
  std::vector<bst_uint> instance_id;
  fvalues.reserve(info.num_col * info.num_row);
  instance_id.reserve(info.num_col * info.num_row);
  feature_id.reserve(info.num_col * info.num_row);

  dmlc::DataIter<ColBatch> *iter = fmat.ColIterator();

  while (iter->Next()) {
    const ColBatch &batch = iter->Value();

    for (int i = 0; i < batch.size; i++) {
      const ColBatch::Inst &col = batch[i];

      for (const ColBatch::Entry *it = col.data; it != col.data + col.length;
           it++) {
        bst_uint inst_id = it->index;
        if (subsample < 1.0) {
          if (row_flags[inst_id]) {
            fvalues.push_back(it->fvalue);
            instance_id.push_back(inst_id);
            feature_id.push_back(i);
          }
        } else {
          fvalues.push_back(it->fvalue);
          instance_id.push_back(inst_id);
          feature_id.push_back(i);
        }
      }
      foffsets.push_back(fvalues.size());
    }
  }

  t.printElapsed("dmatrix");
  t.reset();
  gpu_data->Init(fvalues, foffsets, instance_id, feature_id, gpair,
                 info.num_row, info.num_col, param.max_depth, param);

  t.printElapsed("gpu init");
}

void GPUBuilder::InitFirstNode() {
  // Build the root node on the CPU and copy to device
  gpu_gpair sum_gradients =
      thrust::reduce(gpu_data->gpair.tbegin(), gpu_data->gpair.tend(),
                     gpu_gpair(0, 0), hipcub::Sum());

  Node tmp = Node(
      sum_gradients,
      CalcGain(gpu_data->param, sum_gradients.grad(), sum_gradients.hess()),
      CalcWeight(gpu_data->param, sum_gradients.grad(), sum_gradients.hess()));

  thrust::copy_n(&tmp, 1, gpu_data->nodes.tbegin());
}

enum NodeType {
  NODE = 0,
  LEAF = 1,
  UNUSED = 2,
};

// Recursively label node types
void flag_nodes(const thrust::host_vector<Node> &nodes,
                std::vector<NodeType> *node_flags, int nid, NodeType type) {
  if (nid >= nodes.size() || type == UNUSED) {
    return;
  }

  const Node &n = nodes[nid];

  // Current node and all children are valid
  if (n.split.loss_chg > rt_eps) {
    (*node_flags)[nid] = NODE;
    flag_nodes(nodes, node_flags, nid * 2 + 1, NODE);
    flag_nodes(nodes, node_flags, nid * 2 + 2, NODE);
  } else {
    // Current node is leaf, therefore is valid but all children are invalid
    (*node_flags)[nid] = LEAF;
    flag_nodes(nodes, node_flags, nid * 2 + 1, UNUSED);
    flag_nodes(nodes, node_flags, nid * 2 + 2, UNUSED);
  }
}

// Copy gpu dense representation of tree to xgboost sparse representation
void GPUBuilder::CopyTree(RegTree &tree) {
  std::vector<Node> h_nodes = gpu_data->nodes.as_vector();
  std::vector<NodeType> node_flags(h_nodes.size(), UNUSED);
  flag_nodes(h_nodes, &node_flags, 0, NODE);

  int nid = 0;
  for (int gpu_nid = 0; gpu_nid < h_nodes.size(); gpu_nid++) {
    NodeType flag = node_flags[gpu_nid];
    const Node &n = h_nodes[gpu_nid];
    if (flag == NODE) {
      tree.AddChilds(nid);
      tree[nid].set_split(n.split.findex, n.split.fvalue, n.split.missing_left);
      tree.stat(nid).loss_chg = n.split.loss_chg;
      tree.stat(nid).base_weight = n.weight;
      tree.stat(nid).sum_hess = n.sum_gradients.hess();
      tree[tree[nid].cleft()].set_leaf(0);
      tree[tree[nid].cright()].set_leaf(0);
      nid++;
    } else if (flag == LEAF) {
      tree[nid].set_leaf(n.weight * param.learning_rate);
      tree.stat(nid).sum_hess = n.sum_gradients.hess();
      nid++;
    }
  }
}
}  // namespace tree
}  // namespace xgboost
