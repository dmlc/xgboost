#include "hip/hip_runtime.h"
/*!
 * Copyright 2017 XGBoost contributors
 */
#include <xgboost/tree_updater.h>
#include <utility>
#include <vector>
#include "../../../src/common/sync.h"
#include "../../../src/tree/param.h"
#include "exact/fused_scan_reduce_by_key.cuh"

namespace xgboost {
namespace tree {

DMLC_REGISTRY_FILE_TAG(updater_gpu);

/**
 * @struct ExactSplitCandidate node.cuh
 * @brief Abstraction of a possible split in the decision tree
 */
struct ExactSplitCandidate {
  /** the optimal gain score for this node */
  float score;
  /** index where to split in the DMatrix */
  int index;

  HOST_DEV_INLINE ExactSplitCandidate() : score(-FLT_MAX), index(INT_MAX) {}

  /**
   * @brief Whether the split info is valid to be used to create a new child
   * @param minSplitLoss minimum score above which decision to split is made
   * @return true if splittable, else false
   */
  HOST_DEV_INLINE bool isSplittable(float minSplitLoss) const {
    return ((score >= minSplitLoss) && (index != INT_MAX));
  }
};

/**
 * @enum ArgMaxByKeyAlgo best_split_evaluation.cuh
 * @brief Help decide which algorithm to use for multi-argmax operation
 */
enum ArgMaxByKeyAlgo {
  /** simplest, use gmem-atomics for all updates */
  ABK_GMEM = 0,
  /** use smem-atomics for updates (when number of keys are less) */
  ABK_SMEM
};

/** max depth until which to use shared mem based atomics for argmax */
static const int MAX_ABK_LEVELS = 3;

HOST_DEV_INLINE ExactSplitCandidate maxSplit(ExactSplitCandidate a,
                                             ExactSplitCandidate b) {
  ExactSplitCandidate out;
  if (a.score < b.score) {
    out.score = b.score;
    out.index = b.index;
  } else if (a.score == b.score) {
    out.score = a.score;
    out.index = (a.index < b.index) ? a.index : b.index;
  } else {
    out.score = a.score;
    out.index = a.index;
  }
  return out;
}

DEV_INLINE void atomicArgMax(ExactSplitCandidate* address,
                             ExactSplitCandidate val) {
  unsigned long long* intAddress = (unsigned long long*)address;
  unsigned long long old = *intAddress;
  unsigned long long assumed;
  do {
    assumed = old;
    ExactSplitCandidate res = maxSplit(val, *(ExactSplitCandidate*)&assumed);
    old = atomicCAS(intAddress, assumed, *(uint64_t*)&res);
  } while (assumed != old);
}

DEV_INLINE void argMaxWithAtomics(
    int id, ExactSplitCandidate* nodeSplits, const bst_gpair* gradScans,
    const bst_gpair* gradSums, const float* vals, const int* colIds,
    const node_id_t* nodeAssigns, const DeviceDenseNode* nodes, int nUniqKeys,
    node_id_t nodeStart, int len, const GPUTrainingParam& param) {
  int nodeId = nodeAssigns[id];
  ///@todo: this is really a bad check! but will be fixed when we move
  ///   to key-based reduction
  if ((id == 0) ||
      !((nodeId == nodeAssigns[id - 1]) && (colIds[id] == colIds[id - 1]) &&
        (vals[id] == vals[id - 1]))) {
    if (nodeId != UNUSED_NODE) {
      int sumId = abs2uniqKey(id, nodeAssigns, colIds, nodeStart, nUniqKeys);
      bst_gpair colSum = gradSums[sumId];
      int uid = nodeId - nodeStart;
      DeviceDenseNode n = nodes[nodeId];
      bst_gpair parentSum = n.sum_gradients;
      float parentGain = n.root_gain;
      bool tmp;
      ExactSplitCandidate s;
      bst_gpair missing = parentSum - colSum;
      s.score = loss_chg_missing(gradScans[id], missing, parentSum, parentGain,
                                 param, tmp);
      s.index = id;
      atomicArgMax(nodeSplits + uid, s);
    }  // end if nodeId != UNUSED_NODE
  }    // end if id == 0 ...
}

__global__ void atomicArgMaxByKeyGmem(
    ExactSplitCandidate* nodeSplits, const bst_gpair* gradScans,
    const bst_gpair* gradSums, const float* vals, const int* colIds,
    const node_id_t* nodeAssigns, const DeviceDenseNode* nodes, int nUniqKeys,
    node_id_t nodeStart, int len, const TrainParam param) {
  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  const int stride = blockDim.x * gridDim.x;
  for (; id < len; id += stride) {
    argMaxWithAtomics(id, nodeSplits, gradScans, gradSums, vals, colIds,
                      nodeAssigns, nodes, nUniqKeys, nodeStart, len,
                      GPUTrainingParam(param));
  }
}

__global__ void atomicArgMaxByKeySmem(
    ExactSplitCandidate* nodeSplits, const bst_gpair* gradScans,
    const bst_gpair* gradSums, const float* vals, const int* colIds,
    const node_id_t* nodeAssigns, const DeviceDenseNode* nodes, int nUniqKeys,
    node_id_t nodeStart, int len, const TrainParam param) {
  extern __shared__ char sArr[];
  ExactSplitCandidate* sNodeSplits =
      reinterpret_cast<ExactSplitCandidate*>(sArr);
  int tid = threadIdx.x;
  ExactSplitCandidate defVal;
#pragma unroll 1
  for (int i = tid; i < nUniqKeys; i += blockDim.x) {
    sNodeSplits[i] = defVal;
  }
  __syncthreads();
  int id = tid + (blockIdx.x * blockDim.x);
  const int stride = blockDim.x * gridDim.x;
  for (; id < len; id += stride) {
    argMaxWithAtomics(id, sNodeSplits, gradScans, gradSums, vals, colIds,
                      nodeAssigns, nodes, nUniqKeys, nodeStart, len, param);
  }
  __syncthreads();
  for (int i = tid; i < nUniqKeys; i += blockDim.x) {
    ExactSplitCandidate s = sNodeSplits[i];
    atomicArgMax(nodeSplits + i, s);
  }
}

/**
 * @brief Performs argmax_by_key functionality but for cases when keys need not
 *  occur contiguously
 * @param nodeSplits will contain information on best split for each node
 * @param gradScans exclusive sum on sorted segments for each col
 * @param gradSums gradient sum for each column in DMatrix based on to node-ids
 * @param vals feature values
 * @param colIds column index for each element in the feature values array
 * @param nodeAssigns node-id assignments to each element in DMatrix
 * @param nodes pointer to all nodes for this tree in BFS order
 * @param nUniqKeys number of unique node-ids in this level
 * @param nodeStart start index of the node-ids in this level
 * @param len number of elements
 * @param param training parameters
 * @param algo which algorithm to use for argmax_by_key
 */
template <int BLKDIM = 256, int ITEMS_PER_THREAD = 4>
void argMaxByKey(ExactSplitCandidate* nodeSplits, const bst_gpair* gradScans,
                 const bst_gpair* gradSums, const float* vals,
                 const int* colIds, const node_id_t* nodeAssigns,
                 const DeviceDenseNode* nodes, int nUniqKeys,
                 node_id_t nodeStart, int len, const TrainParam param,
                 ArgMaxByKeyAlgo algo) {
  fillConst<ExactSplitCandidate, BLKDIM, ITEMS_PER_THREAD>(
      dh::get_device_idx(param.gpu_id), nodeSplits, nUniqKeys,
      ExactSplitCandidate());
  int nBlks = dh::div_round_up(len, ITEMS_PER_THREAD * BLKDIM);
  switch (algo) {
    case ABK_GMEM:
      atomicArgMaxByKeyGmem<<<nBlks, BLKDIM>>>(
          nodeSplits, gradScans, gradSums, vals, colIds, nodeAssigns, nodes,
          nUniqKeys, nodeStart, len, param);
      break;
    case ABK_SMEM:
      atomicArgMaxByKeySmem<<<nBlks, BLKDIM,
                              sizeof(ExactSplitCandidate) * nUniqKeys>>>(
          nodeSplits, gradScans, gradSums, vals, colIds, nodeAssigns, nodes,
          nUniqKeys, nodeStart, len, param);
      break;
    default:
      throw std::runtime_error("argMaxByKey: Bad algo passed!");
  }
}

__global__ void assignColIds(int* colIds, const int* colOffsets) {
  int myId = blockIdx.x;
  int start = colOffsets[myId];
  int end = colOffsets[myId + 1];
  for (int id = start + threadIdx.x; id < end; id += blockDim.x) {
    colIds[id] = myId;
  }
}

__global__ void fillDefaultNodeIds(node_id_t* nodeIdsPerInst,
                                   const DeviceDenseNode* nodes, int nRows) {
  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  if (id >= nRows) {
    return;
  }
  // if this element belongs to none of the currently active node-id's
  node_id_t nId = nodeIdsPerInst[id];
  if (nId == UNUSED_NODE) {
    return;
  }
  const DeviceDenseNode n = nodes[nId];
  node_id_t result;
  if (n.IsLeaf() || n.IsUnused()) {
    result = UNUSED_NODE;
  } else if (n.dir == LeftDir) {
    result = (2 * n.idx) + 1;
  } else {
    result = (2 * n.idx) + 2;
  }
  nodeIdsPerInst[id] = result;
}

__global__ void assignNodeIds(node_id_t* nodeIdsPerInst, int* nodeLocations,
                              const node_id_t* nodeIds, const int* instId,
                              const DeviceDenseNode* nodes,
                              const int* colOffsets, const float* vals,
                              int nVals, int nCols) {
  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  const int stride = blockDim.x * gridDim.x;
  for (; id < nVals; id += stride) {
    // fusing generation of indices for node locations
    nodeLocations[id] = id;
    // using nodeIds here since the previous kernel would have updated
    // the nodeIdsPerInst with all default assignments
    int nId = nodeIds[id];
    // if this element belongs to none of the currently active node-id's
    if (nId != UNUSED_NODE) {
      const DeviceDenseNode n = nodes[nId];
      int colId = n.fidx;
      // printf("nid=%d colId=%d id=%d\n", nId, colId, id);
      int start = colOffsets[colId];
      int end = colOffsets[colId + 1];
      ///@todo: too much wasteful threads!!
      if ((id >= start) && (id < end) && !(n.IsLeaf() || n.IsUnused())) {
        node_id_t result = (2 * n.idx) + 1 + (vals[id] >= n.fvalue);
        nodeIdsPerInst[instId[id]] = result;
      }
    }
  }
}

__global__ void markLeavesKernel(DeviceDenseNode* nodes, int len) {
  int id = (blockIdx.x * blockDim.x) + threadIdx.x;
  if ((id < len) && !nodes[id].IsUnused()) {
    int lid = (id << 1) + 1;
    int rid = (id << 1) + 2;
    if ((lid >= len) || (rid >= len)) {
      nodes[id].root_gain = -FLT_MAX;  // bottom-most nodes
    } else if (nodes[lid].IsUnused() && nodes[rid].IsUnused()) {
      nodes[id].root_gain = -FLT_MAX;  // unused child nodes
    }
  }
}

class GPUMaker : public TreeUpdater {
 protected:
  TrainParam param;
  /** whether we have initialized memory already (so as not to repeat!) */
  bool allocated;
  /** feature values stored in column-major compressed format */
  dh::dvec2<float> vals;
  dh::dvec<float> vals_cached;
  /** corresponding instance id's of these featutre values */
  dh::dvec2<int> instIds;
  dh::dvec<int> instIds_cached;
  /** column offsets for these feature values */
  dh::dvec<int> colOffsets;
  dh::dvec<bst_gpair> gradsInst;
  dh::dvec2<node_id_t> nodeAssigns;
  dh::dvec2<int> nodeLocations;
  dh::dvec<DeviceDenseNode> nodes;
  dh::dvec<node_id_t> nodeAssignsPerInst;
  dh::dvec<bst_gpair> gradSums;
  dh::dvec<bst_gpair> gradScans;
  dh::dvec<ExactSplitCandidate> nodeSplits;
  int nVals;
  int nRows;
  int nCols;
  int maxNodes;
  int maxLeaves;
  dh::CubMemory tmp_mem;
  dh::dvec<bst_gpair> tmpScanGradBuff;
  dh::dvec<int> tmpScanKeyBuff;
  dh::dvec<int> colIds;
  dh::bulk_allocator<dh::memory_type::DEVICE> ba;

 public:
  GPUMaker() : allocated(false) {}
  ~GPUMaker() {}

  void Init(
      const std::vector<std::pair<std::string, std::string>>& args) override {
    param.InitAllowUnknown(args);
    maxNodes = (1 << (param.max_depth + 1)) - 1;
    maxLeaves = 1 << param.max_depth;
  }

  void Update(const std::vector<bst_gpair>& gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) override {
    GradStats::CheckInfo(dmat->info());
    // rescale learning rate according to size of trees
    float lr = param.learning_rate;
    param.learning_rate = lr / trees.size();

    try {
      // build tree
      for (size_t i = 0; i < trees.size(); ++i) {
        UpdateTree(gpair, dmat, trees[i]);
      }
    } catch (const std::exception& e) {
      LOG(FATAL) << "GPU plugin exception: " << e.what() << std::endl;
    }
    param.learning_rate = lr;
  }
  /// @note: Update should be only after Init!!
  void UpdateTree(const std::vector<bst_gpair>& gpair, DMatrix* hMat,
                  RegTree* hTree) {
    if (!allocated) {
      setupOneTimeData(*hMat);
    }
    for (int i = 0; i < param.max_depth; ++i) {
      if (i == 0) {
        // make sure to start on a fresh tree with sorted values!
        vals.current_dvec() = vals_cached;
        instIds.current_dvec() = instIds_cached;
        transferGrads(gpair);
      }
      int nNodes = 1 << i;
      node_id_t nodeStart = nNodes - 1;
      initNodeData(i, nodeStart, nNodes);
      findSplit(i, nodeStart, nNodes);
    }
    // mark all the used nodes with unused children as leaf nodes
    markLeaves();
    dense2sparse_tree(hTree, nodes, param);
  }
    // split2node(nodes.data(), nodeSplits.data(), gradScans.data(),
    //           gradSums.data(), vals.current(), colIds.data(),
    //           colOffsets.data(), nodeAssigns.current(), nNodes, nodeStart,
    //           nCols, param);

//__global__ void split2nodeKernel(
//    DeviceDenseNode* nodes, const ExactSplitCandidate* nodeSplits,
//    const bst_gpair* gradScans, const bst_gpair* gradSums, const float* vals,
//    const int* colIds, const int* colOffsets, const node_id_t* nodeAssigns,
//    int nUniqKeys, node_id_t nodeStart, int nCols, const TrainParam param) {

  void split2node(int nNodes, node_id_t nodeStart) {
    auto d_nodes = nodes.data();
    auto d_gradScans = gradScans.data();
    auto d_gradSums = gradSums.data();
    auto d_nodeAssigns = nodeAssigns.current();
    auto d_colIds = colIds.data();
    auto d_vals = vals.current();
    auto d_nodeSplits = nodeSplits.data();
    int nUniqKeys = nNodes;
    float min_split_loss = param.min_split_loss;
    auto gpu_param = GPUTrainingParam(param);

    dh::launch_n(param.gpu_id, nNodes, [=] __device__(int uid) {
      int absNodeId = uid + nodeStart;
      ExactSplitCandidate s = d_nodeSplits[uid];
      if (s.isSplittable(min_split_loss)) {
        int idx = s.index;
        int nodeInstId =
            abs2uniqKey(idx, d_nodeAssigns, d_colIds, nodeStart, nUniqKeys);
        bool missingLeft = true;
        const DeviceDenseNode& n = d_nodes[absNodeId];
        bst_gpair gradScan = d_gradScans[idx];
        bst_gpair gradSum = d_gradSums[nodeInstId];
        float thresh = d_vals[idx];
        int colId = d_colIds[idx];
        // get the default direction for the current node
        bst_gpair missing = n.sum_gradients - gradSum;
        loss_chg_missing(gradScan, missing, n.sum_gradients, n.root_gain, gpu_param,
                         missingLeft);
        // get the score/weight/id/gradSum for left and right child nodes
        bst_gpair lGradSum = missingLeft ? gradScan + missing : gradScan;
        bst_gpair rGradSum = n.sum_gradients - lGradSum;

        // Create children
        d_nodes[left_child_nidx(absNodeId)] =
            DeviceDenseNode(lGradSum, left_child_nidx(absNodeId), gpu_param);
        d_nodes[right_child_nidx(absNodeId)] =
            DeviceDenseNode(rGradSum, right_child_nidx(absNodeId), gpu_param);
        // Set split for parent
        d_nodes[absNodeId].SetSplit(thresh, colId,
                                  missingLeft ? LeftDir : RightDir);
      } else {
        // cannot be split further, so this node is a leaf!
        d_nodes[absNodeId].root_gain = -FLT_MAX;
      }

    });
  }

  void findSplit(int level, node_id_t nodeStart, int nNodes) {
    reduceScanByKey(gradSums.data(), gradScans.data(), gradsInst.data(),
                    instIds.current(), nodeAssigns.current(), nVals, nNodes,
                    nCols, tmpScanGradBuff.data(), tmpScanKeyBuff.data(),
                    colIds.data(), nodeStart);
    argMaxByKey(nodeSplits.data(), gradScans.data(), gradSums.data(),
                vals.current(), colIds.data(), nodeAssigns.current(),
                nodes.data(), nNodes, nodeStart, nVals, param,
                level <= MAX_ABK_LEVELS ? ABK_SMEM : ABK_GMEM);
    // split2node(nodes.data(), nodeSplits.data(), gradScans.data(),
    //           gradSums.data(), vals.current(), colIds.data(),
    //           colOffsets.data(), nodeAssigns.current(), nNodes, nodeStart,
    //           nCols, param);
    split2node(nNodes, nodeStart);
  }

  void allocateAllData(int offsetSize) {
    int tmpBuffSize = scanTempBufferSize(nVals);
    ba.allocate(dh::get_device_idx(param.gpu_id), param.silent, &vals, nVals,
                &vals_cached, nVals, &instIds, nVals, &instIds_cached, nVals,
                &colOffsets, offsetSize, &gradsInst, nRows, &nodeAssigns, nVals,
                &nodeLocations, nVals, &nodes, maxNodes, &nodeAssignsPerInst,
                nRows, &gradSums, maxLeaves * nCols, &gradScans, nVals,
                &nodeSplits, maxLeaves, &tmpScanGradBuff, tmpBuffSize,
                &tmpScanKeyBuff, tmpBuffSize, &colIds, nVals);
  }

  void setupOneTimeData(DMatrix& hMat) {
    size_t free_memory = dh::available_memory(dh::get_device_idx(param.gpu_id));
    if (!hMat.SingleColBlock()) {
      throw std::runtime_error("exact::GPUBuilder - must have 1 column block");
    }
    std::vector<float> fval;
    std::vector<int> fId, offset;
    convertToCsc(hMat, fval, fId, offset);
    allocateAllData((int)offset.size());
    transferAndSortData(fval, fId, offset);
    allocated = true;
  }

  void convertToCsc(DMatrix& hMat, std::vector<float>& fval,
                    std::vector<int>& fId, std::vector<int>& offset) {
    MetaInfo info = hMat.info();
    nRows = info.num_row;
    nCols = info.num_col;
    offset.reserve(nCols + 1);
    offset.push_back(0);
    fval.reserve(nCols * nRows);
    fId.reserve(nCols * nRows);
    // in case you end up with a DMatrix having no column access
    // then make sure to enable that before copying the data!
    if (!hMat.HaveColAccess()) {
      const std::vector<bool> enable(nCols, true);
      hMat.InitColAccess(enable, 1, nRows);
    }
    dmlc::DataIter<ColBatch>* iter = hMat.ColIterator();
    iter->BeforeFirst();
    while (iter->Next()) {
      const ColBatch& batch = iter->Value();
      for (int i = 0; i < batch.size; i++) {
        const ColBatch::Inst& col = batch[i];
        for (const ColBatch::Entry* it = col.data; it != col.data + col.length;
             it++) {
          int inst_id = static_cast<int>(it->index);
          fval.push_back(it->fvalue);
          fId.push_back(inst_id);
        }
        offset.push_back(fval.size());
      }
    }
    nVals = fval.size();
  }

  void transferAndSortData(const std::vector<float>& fval,
                           const std::vector<int>& fId,
                           const std::vector<int>& offset) {
    vals.current_dvec() = fval;
    instIds.current_dvec() = fId;
    colOffsets = offset;
    segmentedSort<float, int>(&tmp_mem, &vals, &instIds, nVals, nCols,
                              colOffsets);
    vals_cached = vals.current_dvec();
    instIds_cached = instIds.current_dvec();
    assignColIds<<<nCols, 512>>>(colIds.data(), colOffsets.data());
  }

  void transferGrads(const std::vector<bst_gpair>& gpair) {
    // HACK
    dh::safe_cuda(hipMemcpy(gradsInst.data(), &(gpair[0]),
                             sizeof(bst_gpair) * nRows,
                             hipMemcpyHostToDevice));
    // evaluate the full-grad reduction for the root node
    sumReduction<bst_gpair>(tmp_mem, gradsInst, gradSums, nRows);
  }

  void initNodeData(int level, node_id_t nodeStart, int nNodes) {
    // all instances belong to root node at the beginning!
    if (level == 0) {
      nodes.fill(DeviceDenseNode());
      nodeAssigns.current_dvec().fill(0);
      nodeAssignsPerInst.fill(0);
      // for root node, just update the gradient/score/weight/id info
      // before splitting it! Currently all data is on GPU, hence this
      // stupid little kernel
      auto d_nodes = nodes.data();
      auto d_sums = gradSums.data();
      auto gpu_params = GPUTrainingParam(param);
      dh::launch_n(param.gpu_id, 1, [=] __device__(int idx) {
        d_nodes[0] = DeviceDenseNode(d_sums[0], 0, gpu_params);
      });
    } else {
      const int BlkDim = 256;
      const int ItemsPerThread = 4;
      // assign default node ids first
      int nBlks = dh::div_round_up(nRows, BlkDim);
      fillDefaultNodeIds<<<nBlks, BlkDim>>>(nodeAssignsPerInst.data(),
                                            nodes.data(), nRows);
      // evaluate the correct child indices of non-missing values next
      nBlks = dh::div_round_up(nVals, BlkDim * ItemsPerThread);
      assignNodeIds<<<nBlks, BlkDim>>>(
          nodeAssignsPerInst.data(), nodeLocations.current(),
          nodeAssigns.current(), instIds.current(), nodes.data(),
          colOffsets.data(), vals.current(), nVals, nCols);
      // gather the node assignments across all other columns too
      gather(dh::get_device_idx(param.gpu_id), nodeAssigns.current(),
             nodeAssignsPerInst.data(), instIds.current(), nVals);
      sortKeys(level);
    }
  }

  void sortKeys(int level) {
    // segmented-sort the arrays based on node-id's
    // but we don't need more than level+1 bits for sorting!
    segmentedSort(&tmp_mem, &nodeAssigns, &nodeLocations, nVals, nCols,
                  colOffsets, 0, level + 1);
    gather<float, int>(dh::get_device_idx(param.gpu_id), vals.other(),
                       vals.current(), instIds.other(), instIds.current(),
                       nodeLocations.current(), nVals);
    vals.buff().selector ^= 1;
    instIds.buff().selector ^= 1;
  }

  void markLeaves() {
    const int BlkDim = 128;
    int nBlks = dh::div_round_up(maxNodes, BlkDim);
    markLeavesKernel<<<nBlks, BlkDim>>>(nodes.data(), maxNodes);
  }
};

XGBOOST_REGISTER_TREE_UPDATER(GPUMaker, "grow_gpu")
    .describe("Grow tree with GPU.")
    .set_body([]() { return new GPUMaker(); });

}  // namespace tree
}  // namespace xgboost
