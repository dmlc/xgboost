#include "hip/hip_runtime.h"
/*!
 * Copyright 2017 XGBoost contributors
 */
#include <xgboost/tree_updater.h>
#include <utility>
#include <vector>
#include "../../../src/tree/param.h"
#include "updater_gpu_common.cuh"

namespace xgboost {
namespace tree {

DMLC_REGISTRY_FILE_TAG(updater_gpu);

/**
 * @brief Absolute BFS order IDs to col-wise unique IDs based on user input
 * @param tid the index of the element that this thread should access
 * @param abs the array of absolute IDs
 * @param colIds the array of column IDs for each element
 * @param nodeStart the start of the node ID at this level
 * @param nKeys number of nodes at this level.
 * @return the uniq key
 */

static HOST_DEV_INLINE node_id_t abs2uniqKey(int tid, const node_id_t* abs,
                                const int* colIds, node_id_t nodeStart,
                                int nKeys) {
  int a = abs[tid];
  if (a == UNUSED_NODE) return a;
  return ((a - nodeStart) + (colIds[tid] * nKeys));
}


/**
 * @struct Pair
 * @brief Pair used for key basd scan operations on bst_gpair
 */
struct Pair {
  int key;
  bst_gpair value;
};

/** define a key that's not used at all in the entire boosting process */
static const int NONE_KEY = -100;

/**
 * @brief Allocate temporary buffers needed for scan operations
 * @param tmpScans gradient buffer
 * @param tmpKeys keys buffer
 * @param size number of elements that will be scanned
 */
template <int BLKDIM_L1L3 = 256>
int scanTempBufferSize(int size) {
  int nBlks = dh::div_round_up(size, BLKDIM_L1L3);
  return nBlks;
}

struct AddByKey {
  template <typename T>
  HOST_DEV_INLINE T operator()(const T& first, const T& second) const {
    T result;
    if (first.key == second.key) {
      result.key = first.key;
      result.value = first.value + second.value;
    } else {
      result.key = second.key;
      result.value = second.value;
    }
    return result;
  }
};

/**
 * @brief Gradient value getter function
 * @param id the index into the vals or instIds array to which to fetch
 * @param vals the gradient value buffer
 * @param instIds instance index buffer
 * @return the expected gradient value
 */
HOST_DEV_INLINE bst_gpair get(int id, const bst_gpair* vals,
                              const int* instIds) {
  id = instIds[id];
  return vals[id];
}

template <int BLKDIM_L1L3>
__global__ void cubScanByKeyL1(bst_gpair* scans, const bst_gpair* vals,
                               const int* instIds, bst_gpair* mScans,
                               int* mKeys, const node_id_t* keys, int nUniqKeys,
                               const int* colIds, node_id_t nodeStart,
                               const int size) {
  Pair rootPair = {NONE_KEY, bst_gpair(0.f, 0.f)};
  int myKey;
  bst_gpair myValue;
  typedef hipcub::BlockScan<Pair, BLKDIM_L1L3> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  Pair threadData;
  int tid = blockIdx.x * BLKDIM_L1L3 + threadIdx.x;
  if (tid < size) {
    myKey = abs2uniqKey(tid, keys, colIds, nodeStart, nUniqKeys);
    myValue = get(tid, vals, instIds);
  } else {
    myKey = NONE_KEY;
    myValue = 0.f;
  }
  threadData.key = myKey;
  threadData.value = myValue;
  // get previous key, especially needed for the last thread in this block
  // in order to pass on the partial scan values.
  // this statement MUST appear before the checks below!
  // else, the result of this shuffle operation will be undefined
  int previousKey = __shfl_up(myKey, 1);
  // Collectively compute the block-wide exclusive prefix sum
  BlockScan(temp_storage)
      .ExclusiveScan(threadData, threadData, rootPair, AddByKey());
  if (tid < size) {
    scans[tid] = threadData.value;
  } else {
    return;
  }
  if (threadIdx.x == BLKDIM_L1L3 - 1) {
    threadData.value =
        (myKey == previousKey) ? threadData.value : bst_gpair(0.0f, 0.0f);
    mKeys[blockIdx.x] = myKey;
    mScans[blockIdx.x] = threadData.value + myValue;
  }
}

template <int BLKSIZE>
__global__ void cubScanByKeyL2(bst_gpair* mScans, int* mKeys, int mLength) {
  typedef hipcub::BlockScan<Pair, BLKSIZE, hipcub::BLOCK_SCAN_WARP_SCANS> BlockScan;
  Pair threadData;
  __shared__ typename BlockScan::TempStorage temp_storage;
  for (int i = threadIdx.x; i < mLength; i += BLKSIZE - 1) {
    threadData.key = mKeys[i];
    threadData.value = mScans[i];
    BlockScan(temp_storage).InclusiveScan(threadData, threadData, AddByKey());
    mScans[i] = threadData.value;
    __syncthreads();
  }
}

template <int BLKDIM_L1L3>
__global__ void cubScanByKeyL3(bst_gpair* sums, bst_gpair* scans,
                               const bst_gpair* vals, const int* instIds,
                               const bst_gpair* mScans, const int* mKeys,
                               const node_id_t* keys, int nUniqKeys,
                               const int* colIds, node_id_t nodeStart,
                               const int size) {
  int relId = threadIdx.x;
  int tid = (blockIdx.x * BLKDIM_L1L3) + relId;
  // to avoid the following warning from nvcc:
  //   __shared__ memory variable with non-empty constructor or destructor
  //     (potential race between threads)
  __shared__ char gradBuff[sizeof(bst_gpair)];
  __shared__ int s_mKeys;
  bst_gpair* s_mScans = reinterpret_cast<bst_gpair*>(gradBuff);
  if (tid >= size) return;
  // cache block-wide partial scan info
  if (relId == 0) {
    s_mKeys = (blockIdx.x > 0) ? mKeys[blockIdx.x - 1] : NONE_KEY;
    s_mScans[0] = (blockIdx.x > 0) ? mScans[blockIdx.x - 1] : bst_gpair();
  }
  int myKey = abs2uniqKey(tid, keys, colIds, nodeStart, nUniqKeys);
  int previousKey =
      tid == 0 ? NONE_KEY
               : abs2uniqKey(tid - 1, keys, colIds, nodeStart, nUniqKeys);
  bst_gpair myValue = scans[tid];
  __syncthreads();
  if (blockIdx.x > 0 && s_mKeys == previousKey) {
    myValue += s_mScans[0];
  }
  if (tid == size - 1) {
    sums[previousKey] = myValue + get(tid, vals, instIds);
  }
  if ((previousKey != myKey) && (previousKey >= 0)) {
    sums[previousKey] = myValue;
    myValue = bst_gpair(0.0f, 0.0f);
  }
  scans[tid] = myValue;
}

/**
 * @brief Performs fused reduce and scan by key functionality. It is assumed
 * that
 *  the keys occur contiguously!
 * @param sums the output gradient reductions for each element performed
 * key-wise
 * @param scans the output gradient scans for each element performed key-wise
 * @param vals the gradients evaluated for each observation.
 * @param instIds instance ids for each element
 * @param keys keys to be used to segment the reductions. They need not occur
 *  contiguously in contrast to scan_by_key. Currently, we need one key per
 *  value in the 'vals' array.
 * @param size number of elements in the 'vals' array
 * @param nUniqKeys max number of uniq keys found per column
 * @param nCols number of columns
 * @param tmpScans temporary scan buffer needed for cub-pyramid algo
 * @param tmpKeys temporary key buffer needed for cub-pyramid algo
 * @param colIds column indices for each element in the array
 * @param nodeStart index of the leftmost node in the current level
 */
template <int BLKDIM_L1L3 = 256, int BLKDIM_L2 = 512>
void reduceScanByKey(bst_gpair* sums, bst_gpair* scans, const bst_gpair* vals,
                     const int* instIds, const node_id_t* keys, int size,
                     int nUniqKeys, int nCols, bst_gpair* tmpScans,
                     int* tmpKeys, const int* colIds, node_id_t nodeStart) {
  int nBlks = dh::div_round_up(size, BLKDIM_L1L3);
  hipMemset(sums, 0, nUniqKeys * nCols * sizeof(bst_gpair));
  cubScanByKeyL1<BLKDIM_L1L3>
      <<<nBlks, BLKDIM_L1L3>>>(scans, vals, instIds, tmpScans, tmpKeys, keys,
                               nUniqKeys, colIds, nodeStart, size);
  cubScanByKeyL2<BLKDIM_L2><<<1, BLKDIM_L2>>>(tmpScans, tmpKeys, nBlks);
  cubScanByKeyL3<BLKDIM_L1L3>
      <<<nBlks, BLKDIM_L1L3>>>(sums, scans, vals, instIds, tmpScans, tmpKeys,
                               keys, nUniqKeys, colIds, nodeStart, size);
}

/**
 * @struct ExactSplitCandidate
 * @brief Abstraction of a possible split in the decision tree
 */
struct ExactSplitCandidate {
  /** the optimal gain score for this node */
  float score;
  /** index where to split in the DMatrix */
  int index;

  HOST_DEV_INLINE ExactSplitCandidate() : score(-FLT_MAX), index(INT_MAX) {}

  /**
   * @brief Whether the split info is valid to be used to create a new child
   * @param minSplitLoss minimum score above which decision to split is made
   * @return true if splittable, else false
   */
  HOST_DEV_INLINE bool isSplittable(float minSplitLoss) const {
    return ((score >= minSplitLoss) && (index != INT_MAX));
  }
};

/**
 * @enum ArgMaxByKeyAlgo best_split_evaluation.cuh
 * @brief Help decide which algorithm to use for multi-argmax operation
 */
enum ArgMaxByKeyAlgo {
  /** simplest, use gmem-atomics for all updates */
  ABK_GMEM = 0,
  /** use smem-atomics for updates (when number of keys are less) */
  ABK_SMEM
};

/** max depth until which to use shared mem based atomics for argmax */
static const int MAX_ABK_LEVELS = 3;

HOST_DEV_INLINE ExactSplitCandidate maxSplit(ExactSplitCandidate a,
                                             ExactSplitCandidate b) {
  ExactSplitCandidate out;
  if (a.score < b.score) {
    out.score = b.score;
    out.index = b.index;
  } else if (a.score == b.score) {
    out.score = a.score;
    out.index = (a.index < b.index) ? a.index : b.index;
  } else {
    out.score = a.score;
    out.index = a.index;
  }
  return out;
}

DEV_INLINE void atomicArgMax(ExactSplitCandidate* address,
                             ExactSplitCandidate val) {
  unsigned long long* intAddress = (unsigned long long*)address;  // NOLINT
  unsigned long long old = *intAddress;                           // NOLINT
  unsigned long long assumed;                                     // NOLINT
  do {
    assumed = old;
    ExactSplitCandidate res =
        maxSplit(val, *reinterpret_cast<ExactSplitCandidate*>(&assumed));
    old = atomicCAS(intAddress, assumed, *reinterpret_cast<uint64_t*>(&res));
  } while (assumed != old);
}

DEV_INLINE void argMaxWithAtomics(
    int id, ExactSplitCandidate* nodeSplits, const bst_gpair* gradScans,
    const bst_gpair* gradSums, const float* vals, const int* colIds,
    const node_id_t* nodeAssigns, const DeviceDenseNode* nodes, int nUniqKeys,
    node_id_t nodeStart, int len, const GPUTrainingParam& param) {
  int nodeId = nodeAssigns[id];
  // @todo: this is really a bad check! but will be fixed when we move
  //  to key-based reduction
  if ((id == 0) ||
      !((nodeId == nodeAssigns[id - 1]) && (colIds[id] == colIds[id - 1]) &&
        (vals[id] == vals[id - 1]))) {
    if (nodeId != UNUSED_NODE) {
      int sumId = abs2uniqKey(id, nodeAssigns, colIds, nodeStart, nUniqKeys);
      bst_gpair colSum = gradSums[sumId];
      int uid = nodeId - nodeStart;
      DeviceDenseNode n = nodes[nodeId];
      bst_gpair parentSum = n.sum_gradients;
      float parentGain = n.root_gain;
      bool tmp;
      ExactSplitCandidate s;
      bst_gpair missing = parentSum - colSum;
      s.score = loss_chg_missing(gradScans[id], missing, parentSum, parentGain,
                                 param, tmp);
      s.index = id;
      atomicArgMax(nodeSplits + uid, s);
    }  // end if nodeId != UNUSED_NODE
  }    // end if id == 0 ...
}

__global__ void atomicArgMaxByKeyGmem(
    ExactSplitCandidate* nodeSplits, const bst_gpair* gradScans,
    const bst_gpair* gradSums, const float* vals, const int* colIds,
    const node_id_t* nodeAssigns, const DeviceDenseNode* nodes, int nUniqKeys,
    node_id_t nodeStart, int len, const TrainParam param) {
  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  const int stride = blockDim.x * gridDim.x;
  for (; id < len; id += stride) {
    argMaxWithAtomics(id, nodeSplits, gradScans, gradSums, vals, colIds,
                      nodeAssigns, nodes, nUniqKeys, nodeStart, len,
                      GPUTrainingParam(param));
  }
}

__global__ void atomicArgMaxByKeySmem(
    ExactSplitCandidate* nodeSplits, const bst_gpair* gradScans,
    const bst_gpair* gradSums, const float* vals, const int* colIds,
    const node_id_t* nodeAssigns, const DeviceDenseNode* nodes, int nUniqKeys,
    node_id_t nodeStart, int len, const TrainParam param) {
  extern __shared__ char sArr[];
  ExactSplitCandidate* sNodeSplits =
      reinterpret_cast<ExactSplitCandidate*>(sArr);
  int tid = threadIdx.x;
  ExactSplitCandidate defVal;
#pragma unroll 1
  for (int i = tid; i < nUniqKeys; i += blockDim.x) {
    sNodeSplits[i] = defVal;
  }
  __syncthreads();
  int id = tid + (blockIdx.x * blockDim.x);
  const int stride = blockDim.x * gridDim.x;
  for (; id < len; id += stride) {
    argMaxWithAtomics(id, sNodeSplits, gradScans, gradSums, vals, colIds,
                      nodeAssigns, nodes, nUniqKeys, nodeStart, len, param);
  }
  __syncthreads();
  for (int i = tid; i < nUniqKeys; i += blockDim.x) {
    ExactSplitCandidate s = sNodeSplits[i];
    atomicArgMax(nodeSplits + i, s);
  }
}

/**
 * @brief Performs argmax_by_key functionality but for cases when keys need not
 *  occur contiguously
 * @param nodeSplits will contain information on best split for each node
 * @param gradScans exclusive sum on sorted segments for each col
 * @param gradSums gradient sum for each column in DMatrix based on to node-ids
 * @param vals feature values
 * @param colIds column index for each element in the feature values array
 * @param nodeAssigns node-id assignments to each element in DMatrix
 * @param nodes pointer to all nodes for this tree in BFS order
 * @param nUniqKeys number of unique node-ids in this level
 * @param nodeStart start index of the node-ids in this level
 * @param len number of elements
 * @param param training parameters
 * @param algo which algorithm to use for argmax_by_key
 */
template <int BLKDIM = 256, int ITEMS_PER_THREAD = 4>
void argMaxByKey(ExactSplitCandidate* nodeSplits, const bst_gpair* gradScans,
                 const bst_gpair* gradSums, const float* vals,
                 const int* colIds, const node_id_t* nodeAssigns,
                 const DeviceDenseNode* nodes, int nUniqKeys,
                 node_id_t nodeStart, int len, const TrainParam param,
                 ArgMaxByKeyAlgo algo) {
  dh::fillConst<ExactSplitCandidate, BLKDIM, ITEMS_PER_THREAD>(
      dh::get_device_idx(param.gpu_id), nodeSplits, nUniqKeys,
      ExactSplitCandidate());
  int nBlks = dh::div_round_up(len, ITEMS_PER_THREAD * BLKDIM);
  switch (algo) {
    case ABK_GMEM:
      atomicArgMaxByKeyGmem<<<nBlks, BLKDIM>>>(
          nodeSplits, gradScans, gradSums, vals, colIds, nodeAssigns, nodes,
          nUniqKeys, nodeStart, len, param);
      break;
    case ABK_SMEM:
      atomicArgMaxByKeySmem<<<nBlks, BLKDIM,
                              sizeof(ExactSplitCandidate) * nUniqKeys>>>(
          nodeSplits, gradScans, gradSums, vals, colIds, nodeAssigns, nodes,
          nUniqKeys, nodeStart, len, param);
      break;
    default:
      throw std::runtime_error("argMaxByKey: Bad algo passed!");
  }
}

__global__ void assignColIds(int* colIds, const int* colOffsets) {
  int myId = blockIdx.x;
  int start = colOffsets[myId];
  int end = colOffsets[myId + 1];
  for (int id = start + threadIdx.x; id < end; id += blockDim.x) {
    colIds[id] = myId;
  }
}

__global__ void fillDefaultNodeIds(node_id_t* nodeIdsPerInst,
                                   const DeviceDenseNode* nodes, int nRows) {
  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  if (id >= nRows) {
    return;
  }
  // if this element belongs to none of the currently active node-id's
  node_id_t nId = nodeIdsPerInst[id];
  if (nId == UNUSED_NODE) {
    return;
  }
  const DeviceDenseNode n = nodes[nId];
  node_id_t result;
  if (n.IsLeaf() || n.IsUnused()) {
    result = UNUSED_NODE;
  } else if (n.dir == LeftDir) {
    result = (2 * n.idx) + 1;
  } else {
    result = (2 * n.idx) + 2;
  }
  nodeIdsPerInst[id] = result;
}

__global__ void assignNodeIds(node_id_t* nodeIdsPerInst, int* nodeLocations,
                              const node_id_t* nodeIds, const int* instId,
                              const DeviceDenseNode* nodes,
                              const int* colOffsets, const float* vals,
                              int nVals, int nCols) {
  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  const int stride = blockDim.x * gridDim.x;
  for (; id < nVals; id += stride) {
    // fusing generation of indices for node locations
    nodeLocations[id] = id;
    // using nodeIds here since the previous kernel would have updated
    // the nodeIdsPerInst with all default assignments
    int nId = nodeIds[id];
    // if this element belongs to none of the currently active node-id's
    if (nId != UNUSED_NODE) {
      const DeviceDenseNode n = nodes[nId];
      int colId = n.fidx;
      // printf("nid=%d colId=%d id=%d\n", nId, colId, id);
      int start = colOffsets[colId];
      int end = colOffsets[colId + 1];
      // @todo: too much wasteful threads!!
      if ((id >= start) && (id < end) && !(n.IsLeaf() || n.IsUnused())) {
        node_id_t result = (2 * n.idx) + 1 + (vals[id] >= n.fvalue);
        nodeIdsPerInst[instId[id]] = result;
      }
    }
  }
}

__global__ void markLeavesKernel(DeviceDenseNode* nodes, int len) {
  int id = (blockIdx.x * blockDim.x) + threadIdx.x;
  if ((id < len) && !nodes[id].IsUnused()) {
    int lid = (id << 1) + 1;
    int rid = (id << 1) + 2;
    if ((lid >= len) || (rid >= len)) {
      nodes[id].root_gain = -FLT_MAX;  // bottom-most nodes
    } else if (nodes[lid].IsUnused() && nodes[rid].IsUnused()) {
      nodes[id].root_gain = -FLT_MAX;  // unused child nodes
    }
  }
}

class GPUMaker : public TreeUpdater {
 protected:
  TrainParam param;
  /** whether we have initialized memory already (so as not to repeat!) */
  bool allocated;
  /** feature values stored in column-major compressed format */
  dh::dvec2<float> vals;
  dh::dvec<float> vals_cached;
  /** corresponding instance id's of these featutre values */
  dh::dvec2<int> instIds;
  dh::dvec<int> instIds_cached;
  /** column offsets for these feature values */
  dh::dvec<int> colOffsets;
  dh::dvec<bst_gpair> gradsInst;
  dh::dvec2<node_id_t> nodeAssigns;
  dh::dvec2<int> nodeLocations;
  dh::dvec<DeviceDenseNode> nodes;
  dh::dvec<node_id_t> nodeAssignsPerInst;
  dh::dvec<bst_gpair> gradSums;
  dh::dvec<bst_gpair> gradScans;
  dh::dvec<ExactSplitCandidate> nodeSplits;
  int nVals;
  int nRows;
  int nCols;
  int maxNodes;
  int maxLeaves;
  dh::CubMemory tmp_mem;
  dh::dvec<bst_gpair> tmpScanGradBuff;
  dh::dvec<int> tmpScanKeyBuff;
  dh::dvec<int> colIds;
  dh::bulk_allocator<dh::memory_type::DEVICE> ba;

 public:
  GPUMaker() : allocated(false) {}
  ~GPUMaker() {}

  void Init(
      const std::vector<std::pair<std::string, std::string>>& args) override {
    param.InitAllowUnknown(args);
    maxNodes = (1 << (param.max_depth + 1)) - 1;
    maxLeaves = 1 << param.max_depth;
  }

  void Update(const std::vector<bst_gpair>& gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) override {
    GradStats::CheckInfo(dmat->info());
    // rescale learning rate according to size of trees
    float lr = param.learning_rate;
    param.learning_rate = lr / trees.size();

    try {
      // build tree
      for (size_t i = 0; i < trees.size(); ++i) {
        UpdateTree(gpair, dmat, trees[i]);
      }
    } catch (const std::exception& e) {
      LOG(FATAL) << "GPU plugin exception: " << e.what() << std::endl;
    }
    param.learning_rate = lr;
  }
  /// @note: Update should be only after Init!!
  void UpdateTree(const std::vector<bst_gpair>& gpair, DMatrix* dmat,
                  RegTree* hTree) {
    if (!allocated) {
      setupOneTimeData(dmat);
    }
    for (int i = 0; i < param.max_depth; ++i) {
      if (i == 0) {
        // make sure to start on a fresh tree with sorted values!
        vals.current_dvec() = vals_cached;
        instIds.current_dvec() = instIds_cached;
        transferGrads(gpair);
      }
      int nNodes = 1 << i;
      node_id_t nodeStart = nNodes - 1;
      initNodeData(i, nodeStart, nNodes);
      findSplit(i, nodeStart, nNodes);
    }
    // mark all the used nodes with unused children as leaf nodes
    markLeaves();
    dense2sparse_tree(hTree, nodes, param);
  }

  void split2node(int nNodes, node_id_t nodeStart) {
    auto d_nodes = nodes.data();
    auto d_gradScans = gradScans.data();
    auto d_gradSums = gradSums.data();
    auto d_nodeAssigns = nodeAssigns.current();
    auto d_colIds = colIds.data();
    auto d_vals = vals.current();
    auto d_nodeSplits = nodeSplits.data();
    int nUniqKeys = nNodes;
    float min_split_loss = param.min_split_loss;
    auto gpu_param = GPUTrainingParam(param);

    dh::launch_n(param.gpu_id, nNodes, [=] __device__(int uid) {
      int absNodeId = uid + nodeStart;
      ExactSplitCandidate s = d_nodeSplits[uid];
      if (s.isSplittable(min_split_loss)) {
        int idx = s.index;
        int nodeInstId =
            abs2uniqKey(idx, d_nodeAssigns, d_colIds, nodeStart, nUniqKeys);
        bool missingLeft = true;
        const DeviceDenseNode& n = d_nodes[absNodeId];
        bst_gpair gradScan = d_gradScans[idx];
        bst_gpair gradSum = d_gradSums[nodeInstId];
        float thresh = d_vals[idx];
        int colId = d_colIds[idx];
        // get the default direction for the current node
        bst_gpair missing = n.sum_gradients - gradSum;
        loss_chg_missing(gradScan, missing, n.sum_gradients, n.root_gain,
                         gpu_param, missingLeft);
        // get the score/weight/id/gradSum for left and right child nodes
        bst_gpair lGradSum = missingLeft ? gradScan + missing : gradScan;
        bst_gpair rGradSum = n.sum_gradients - lGradSum;

        // Create children
        d_nodes[left_child_nidx(absNodeId)] =
            DeviceDenseNode(lGradSum, left_child_nidx(absNodeId), gpu_param);
        d_nodes[right_child_nidx(absNodeId)] =
            DeviceDenseNode(rGradSum, right_child_nidx(absNodeId), gpu_param);
        // Set split for parent
        d_nodes[absNodeId].SetSplit(thresh, colId,
                                    missingLeft ? LeftDir : RightDir);
      } else {
        // cannot be split further, so this node is a leaf!
        d_nodes[absNodeId].root_gain = -FLT_MAX;
      }
    });
  }

  void findSplit(int level, node_id_t nodeStart, int nNodes) {
    reduceScanByKey(gradSums.data(), gradScans.data(), gradsInst.data(),
                    instIds.current(), nodeAssigns.current(), nVals, nNodes,
                    nCols, tmpScanGradBuff.data(), tmpScanKeyBuff.data(),
                    colIds.data(), nodeStart);
    argMaxByKey(nodeSplits.data(), gradScans.data(), gradSums.data(),
                vals.current(), colIds.data(), nodeAssigns.current(),
                nodes.data(), nNodes, nodeStart, nVals, param,
                level <= MAX_ABK_LEVELS ? ABK_SMEM : ABK_GMEM);
    split2node(nNodes, nodeStart);
  }

  void allocateAllData(int offsetSize) {
    int tmpBuffSize = scanTempBufferSize(nVals);
    ba.allocate(dh::get_device_idx(param.gpu_id), param.silent, &vals, nVals,
                &vals_cached, nVals, &instIds, nVals, &instIds_cached, nVals,
                &colOffsets, offsetSize, &gradsInst, nRows, &nodeAssigns, nVals,
                &nodeLocations, nVals, &nodes, maxNodes, &nodeAssignsPerInst,
                nRows, &gradSums, maxLeaves * nCols, &gradScans, nVals,
                &nodeSplits, maxLeaves, &tmpScanGradBuff, tmpBuffSize,
                &tmpScanKeyBuff, tmpBuffSize, &colIds, nVals);
  }

  void setupOneTimeData(DMatrix* dmat) {
    size_t free_memory = dh::available_memory(dh::get_device_idx(param.gpu_id));
    if (!dmat->SingleColBlock()) {
      throw std::runtime_error("exact::GPUBuilder - must have 1 column block");
    }
    std::vector<float> fval;
    std::vector<int> fId, offset;
    convertToCsc(dmat, &fval, &fId, &offset);
    allocateAllData(static_cast<int>(offset.size()));
    transferAndSortData(fval, fId, offset);
    allocated = true;
  }

  void convertToCsc(DMatrix* dmat, std::vector<float>* fval,
                    std::vector<int>* fId, std::vector<int>* offset) {
    MetaInfo info = dmat->info();
    nRows = info.num_row;
    nCols = info.num_col;
    offset->reserve(nCols + 1);
    offset->push_back(0);
    fval->reserve(nCols * nRows);
    fId->reserve(nCols * nRows);
    // in case you end up with a DMatrix having no column access
    // then make sure to enable that before copying the data!
    if (!dmat->HaveColAccess()) {
      const std::vector<bool> enable(nCols, true);
      dmat->InitColAccess(enable, 1, nRows);
    }
    dmlc::DataIter<ColBatch>* iter = dmat->ColIterator();
    iter->BeforeFirst();
    while (iter->Next()) {
      const ColBatch& batch = iter->Value();
      for (int i = 0; i < batch.size; i++) {
        const ColBatch::Inst& col = batch[i];
        for (const ColBatch::Entry* it = col.data; it != col.data + col.length;
             it++) {
          int inst_id = static_cast<int>(it->index);
          fval->push_back(it->fvalue);
          fId->push_back(inst_id);
        }
        offset->push_back(fval->size());
      }
    }
    nVals = fval->size();
  }

  void transferAndSortData(const std::vector<float>& fval,
                           const std::vector<int>& fId,
                           const std::vector<int>& offset) {
    vals.current_dvec() = fval;
    instIds.current_dvec() = fId;
    colOffsets = offset;
    dh::segmentedSort<float, int>(&tmp_mem, &vals, &instIds, nVals, nCols,
                              colOffsets);
    vals_cached = vals.current_dvec();
    instIds_cached = instIds.current_dvec();
    assignColIds<<<nCols, 512>>>(colIds.data(), colOffsets.data());
  }

  void transferGrads(const std::vector<bst_gpair>& gpair) {
    // HACK
    dh::safe_cuda(hipMemcpy(gradsInst.data(), &(gpair[0]),
                             sizeof(bst_gpair) * nRows,
                             hipMemcpyHostToDevice));
    // evaluate the full-grad reduction for the root node
    dh::sumReduction<bst_gpair>(tmp_mem, gradsInst, gradSums, nRows);
  }

  void initNodeData(int level, node_id_t nodeStart, int nNodes) {
    // all instances belong to root node at the beginning!
    if (level == 0) {
      nodes.fill(DeviceDenseNode());
      nodeAssigns.current_dvec().fill(0);
      nodeAssignsPerInst.fill(0);
      // for root node, just update the gradient/score/weight/id info
      // before splitting it! Currently all data is on GPU, hence this
      // stupid little kernel
      auto d_nodes = nodes.data();
      auto d_sums = gradSums.data();
      auto gpu_params = GPUTrainingParam(param);
      dh::launch_n(param.gpu_id, 1, [=] __device__(int idx) {
        d_nodes[0] = DeviceDenseNode(d_sums[0], 0, gpu_params);
      });
    } else {
      const int BlkDim = 256;
      const int ItemsPerThread = 4;
      // assign default node ids first
      int nBlks = dh::div_round_up(nRows, BlkDim);
      fillDefaultNodeIds<<<nBlks, BlkDim>>>(nodeAssignsPerInst.data(),
                                            nodes.data(), nRows);
      // evaluate the correct child indices of non-missing values next
      nBlks = dh::div_round_up(nVals, BlkDim * ItemsPerThread);
      assignNodeIds<<<nBlks, BlkDim>>>(
          nodeAssignsPerInst.data(), nodeLocations.current(),
          nodeAssigns.current(), instIds.current(), nodes.data(),
          colOffsets.data(), vals.current(), nVals, nCols);
      // gather the node assignments across all other columns too
      dh::gather(dh::get_device_idx(param.gpu_id), nodeAssigns.current(),
             nodeAssignsPerInst.data(), instIds.current(), nVals);
      sortKeys(level);
    }
  }

  void sortKeys(int level) {
    // segmented-sort the arrays based on node-id's
    // but we don't need more than level+1 bits for sorting!
    segmentedSort(&tmp_mem, &nodeAssigns, &nodeLocations, nVals, nCols,
                  colOffsets, 0, level + 1);
    dh::gather<float, int>(dh::get_device_idx(param.gpu_id), vals.other(),
                       vals.current(), instIds.other(), instIds.current(),
                       nodeLocations.current(), nVals);
    vals.buff().selector ^= 1;
    instIds.buff().selector ^= 1;
  }

  void markLeaves() {
    const int BlkDim = 128;
    int nBlks = dh::div_round_up(maxNodes, BlkDim);
    markLeavesKernel<<<nBlks, BlkDim>>>(nodes.data(), maxNodes);
  }
};

XGBOOST_REGISTER_TREE_UPDATER(GPUMaker, "grow_gpu")
    .describe("Grow tree with GPU.")
    .set_body([]() { return new GPUMaker(); });

}  // namespace tree
}  // namespace xgboost
