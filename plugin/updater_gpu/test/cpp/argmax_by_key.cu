/*
 * Copyright (c) 2017, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "gtest/gtest.h"
#include "../../src/exact/argmax_by_key.cuh"
#include "../../src/exact/gradients.cuh"
#include "../../src/exact/node.cuh"
#include "../../src/exact/loss_functions.cuh"
#include "utils.cuh"


namespace xgboost {
namespace tree {
namespace exact {

TEST(ArgMaxByKey, maxSplit) {
  Split a, b, out;
  a.score = 2.f;
  a.index = 3;
  b.score = 3.f;
  b.index = 4;
  out = maxSplit(a, b);
  EXPECT_FLOAT_EQ(out.score, b.score);
  EXPECT_EQ(out.index, b.index);
  b.score = 2.f;
  b.index = 4;
  out = maxSplit(a, b);
  EXPECT_FLOAT_EQ(out.score, a.score);
  EXPECT_EQ(out.index, a.index);
  b.score = 2.f;
  b.index = 2;
  out = maxSplit(a, b);
  EXPECT_FLOAT_EQ(out.score, a.score);
  EXPECT_EQ(out.index, b.index);
  b.score = 1.f;
  b.index = 1;
  out = maxSplit(a, b);
  EXPECT_FLOAT_EQ(out.score, a.score);
  EXPECT_EQ(out.index, a.index);
}

template <typename node_id_t>
void argMaxTest(ArgMaxByKeyAlgo algo) {
  const int nVals = 1024;
  const int level = 0;
  const int nKeys = 1 << level;
  gpu_gpair* scans = new gpu_gpair[nVals];
  float* vals = new float[nVals];
  int* colIds = new int[nVals];
  scans[0] = gpu_gpair();
  vals[0] = 0.f;
  colIds[0] = 0;
  for (int i = 1; i < nVals; ++i) {
    scans[i].g = scans[i-1].g + (0.1f * 2.f);
    scans[i].h = scans[i-1].h + (0.1f * 2.f);
    vals[i] = static_cast<float>(i) * 0.1f;
    colIds[i] = 0;
  }
  float* dVals;
  allocateAndUpdateOnGpu<float>(dVals, vals, nVals);
  gpu_gpair* dScans;
  allocateAndUpdateOnGpu<gpu_gpair>(dScans, scans, nVals);
  gpu_gpair* sums = new gpu_gpair[nKeys];
  sums[0].g = sums[0].h = (0.1f * 2.f * nVals);
  gpu_gpair* dSums;
  allocateAndUpdateOnGpu<gpu_gpair>(dSums, sums, nKeys);
  int* dColIds;
  allocateAndUpdateOnGpu<int>(dColIds, colIds, nVals);
  Split* splits = new Split[nKeys];
  Split* dSplits;
  allocateOnGpu<Split>(dSplits, nKeys);
  node_id_t* nodeAssigns = new node_id_t[nVals];
  memset(nodeAssigns, 0, sizeof(node_id_t)*nVals);
  node_id_t* dNodeAssigns;
  allocateAndUpdateOnGpu<node_id_t>(dNodeAssigns, nodeAssigns, nVals);
  Node<node_id_t>* nodes = new Node<node_id_t>[nKeys];
  nodes[0].gradSum = sums[0];
  nodes[0].id = 0;
  TrainParam param;
  param.min_child_weight = 0.0f;
  param.reg_alpha = 0.f;
  param.reg_lambda = 2.f;
  param.max_delta_step = 0.f;
  nodes[0].score = CalcGain(param, sums[0].g, sums[0].h);
  Node<node_id_t>* dNodes;
  allocateAndUpdateOnGpu<Node<node_id_t> >(dNodes, nodes, nKeys);
  argMaxByKey<node_id_t>(dSplits, dScans, dSums, dVals, dColIds, dNodeAssigns,
                         dNodes, nKeys, 0, nVals, param, algo);
  updateHostPtr<Split>(splits, dSplits, nKeys);
  EXPECT_FLOAT_EQ(0.f, splits->score);
  EXPECT_EQ(0, splits->index);
  dh::safe_cuda(hipFree(dNodeAssigns));
  delete [] nodeAssigns;
  dh::safe_cuda(hipFree(dSplits));
  delete [] splits;
  dh::safe_cuda(hipFree(dColIds));
  delete [] colIds;
  dh::safe_cuda(hipFree(dSums));
  delete [] sums;
  dh::safe_cuda(hipFree(dVals));
  delete [] vals;
  dh::safe_cuda(hipFree(dScans));
  delete [] scans;
}

TEST(ArgMaxByKey, testOneColGmem) {
  argMaxTest<int16_t>(ABK_GMEM);
}

TEST(ArgMaxByKey, testOneColSmem) {
  argMaxTest<int16_t>(ABK_SMEM);
}

}  // namespace exact
}  // namespace tree
}  // namespace xgboost
