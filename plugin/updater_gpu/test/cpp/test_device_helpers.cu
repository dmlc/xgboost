
/*!
 * Copyright 2017 XGBoost contributors
 */
#include <thrust/device_vector.h>
#include <xgboost/base.h>
#include "../../src/device_helpers.cuh"
#include "gtest/gtest.h"

static const std::vector<int> gidx = {0, 2, 5, 1, 3, 6, 0, 2, 0, 7};
static const std::vector<int> row_ptr = {0, 3, 6, 8, 10};
static const std::vector<int> lbs_seg_output = {0, 0, 0, 1, 1, 1, 2, 2, 3, 3};

thrust::device_vector<int> test_lbs() {
  thrust::device_vector<int> device_gidx = gidx;
  thrust::device_vector<int> device_row_ptr = row_ptr;
  thrust::device_vector<int> device_output_row(gidx.size(), 0);
  auto d_output_row = device_output_row.data();
  dh::CubMemory temp_memory;
  dh::TransformLbs(
      0, &temp_memory, gidx.size(), device_row_ptr.data(), row_ptr.size() - 1,
      [=] __device__(int idx, int ridx) { d_output_row[idx] = ridx; });

  dh::safe_cuda(hipDeviceSynchronize());
  return device_output_row;
}

TEST(lbs, Test) { ASSERT_TRUE(test_lbs() == lbs_seg_output); }
