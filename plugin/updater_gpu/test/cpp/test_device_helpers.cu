
/*!
 * Copyright 2017 XGBoost contributors
 */
#include <thrust/device_vector.h>
#include <xgboost/base.h>
#include "../../src/device_helpers.cuh"
#include "gtest/gtest.h"

void CreateTestData(xgboost::bst_uint num_rows, int max_row_size,
                    thrust::host_vector<int> *row_ptr,
                    thrust::host_vector<xgboost::bst_uint> *rows) {
  row_ptr->resize(num_rows + 1);
  int sum = 0;
  for (int i = 0; i <= num_rows; i++) {
    (*row_ptr)[i] = sum;
    sum += rand() % max_row_size;  // NOLINT

    if (i < num_rows) {
      for (int j = (*row_ptr)[i]; j < sum; j++) {
        (*rows).push_back(i);
      }
    }
  }
}

void SpeedTest() {
  int num_rows = 1000000;
  int max_row_size = 100;
  dh::CubMemory temp_memory;
  thrust::host_vector<int> h_row_ptr;
  thrust::host_vector<xgboost::bst_uint> h_rows;
  CreateTestData(num_rows, max_row_size, &h_row_ptr, &h_rows);
  thrust::device_vector<int> row_ptr = h_row_ptr;
  thrust::device_vector<int> output_row(h_rows.size());
  auto d_output_row = output_row.data();

  dh::Timer t;
  dh::TransformLbs(
      0, &temp_memory, h_rows.size(), dh::raw(row_ptr), row_ptr.size() - 1, false,
      [=] __device__(size_t idx, size_t ridx) { d_output_row[idx] = ridx; });

  dh::safe_cuda(hipDeviceSynchronize());
  double time = t.elapsedSeconds();
  const int mb_size = 1048576;
  size_t size = (sizeof(int) * h_rows.size()) / mb_size;
  printf("size: %llumb, time: %fs, bandwidth: %fmb/s\n", size, time,
         size / time);
}

void TestLbs() {
  srand(17);
  dh::CubMemory temp_memory;

  std::vector<int> test_rows = {4, 100, 1000};
  std::vector<int> test_max_row_sizes = {4, 100, 1300};

  for (auto num_rows : test_rows) {
    for (auto max_row_size : test_max_row_sizes) {
      thrust::host_vector<int> h_row_ptr;
      thrust::host_vector<xgboost::bst_uint> h_rows;
      CreateTestData(num_rows, max_row_size, &h_row_ptr, &h_rows);
      thrust::device_vector<size_t> row_ptr = h_row_ptr;
      thrust::device_vector<int> output_row(h_rows.size());
      auto d_output_row = output_row.data();

      dh::TransformLbs(0, &temp_memory, h_rows.size(), dh::raw(row_ptr),
                       row_ptr.size() - 1, false,
                       [=] __device__(size_t idx, size_t ridx) {
                         d_output_row[idx] = ridx;
                       });

      dh::safe_cuda(hipDeviceSynchronize());
      ASSERT_TRUE(h_rows == output_row);
    }
  }
}
TEST(cub_lbs, Test) { TestLbs(); }
