/**
 * Copyright 2021-2025, XGBoost Contributors
 */
#include <jni.h>
#include <xgboost/c_api.h>

#include "../../../../src/common/cuda_pinned_allocator.h"
#include "../../../../src/common/device_vector.cuh"  // for device_vector
#include "../../../../src/data/array_interface.h"
#include "jvm_utils.h"  // for CheckJvmCall

namespace xgboost::jni {
template <typename T, typename Alloc>
T const *RawPtr(std::vector<T, Alloc> const &data) {
  return data.data();
}

template <typename T, typename Alloc>
T *RawPtr(std::vector<T, Alloc> &data) {
  return data.data();
}

template <typename T>
T const *RawPtr(dh::device_vector<T> const &data) {
  return data.data().get();
}

template <typename T>
T *RawPtr(dh::device_vector<T> &data) {
  return data.data().get();
}

template <typename VCont>
void CopyColumnMask(xgboost::ArrayInterface<1> const &interface, std::vector<Json> const &columns,
                    hipMemcpyKind kind, size_t c, VCont *p_mask, Json *p_out,
                    hipStream_t stream) {
  auto &mask = *p_mask;
  auto &out = *p_out;
  auto size = sizeof(typename VCont::value_type) * interface.n;
  mask.resize(size);
  CHECK(RawPtr(mask));
  CHECK(size);
  CHECK(interface.valid.Data());
  dh::safe_cuda(hipMemcpyAsync(RawPtr(mask), interface.valid.Data(), size, kind, stream));
  auto const &mask_column = columns[c]["mask"];
  out["mask"] = Object();
  std::vector<Json> mask_data{Json{reinterpret_cast<Integer::Int>(RawPtr(mask))},
                              Json{get<Boolean const>(mask_column["data"][1])}};
  out["mask"]["data"] = Array(std::move(mask_data));
  if (get<Array const>(mask_column["shape"]).size() == 2) {
    std::vector<Json> mask_shape{Json{get<Integer const>(mask_column["shape"][0])},
                                 Json{get<Integer const>(mask_column["shape"][1])}};
    out["mask"]["shape"] = Array(std::move(mask_shape));
  } else if (get<Array const>(mask_column["shape"]).size() == 1) {
    std::vector<Json> mask_shape{Json{get<Integer const>(mask_column["shape"][0])}};
    out["mask"]["shape"] = Array(std::move(mask_shape));
  } else {
    LOG(FATAL) << "Invalid shape of mask";
  }
  out["mask"]["typestr"] = String("<t1");
  out["mask"]["version"] = Integer(3);
}

template <typename DCont, typename VCont>
void CopyInterface(std::vector<xgboost::ArrayInterface<1>> &interface_arr,
                   std::vector<Json> const &columns, hipMemcpyKind kind,
                   std::vector<DCont> *p_data, std::vector<VCont> *p_mask,
                   std::vector<xgboost::Json> *p_out, hipStream_t stream) {
  p_data->resize(interface_arr.size());
  p_mask->resize(interface_arr.size());
  p_out->resize(interface_arr.size());
  for (size_t c = 0; c < interface_arr.size(); ++c) {
    auto &interface = interface_arr.at(c);
    size_t element_size = interface.ElementSize();
    size_t size = element_size * interface.n;

    auto &data = (*p_data)[c];
    auto &mask = (*p_mask)[c];
    data.resize(size);
    dh::safe_cuda(hipMemcpyAsync(RawPtr(data), interface.data, size, kind, stream));

    auto &out = (*p_out)[c];
    out = Object();
    std::vector<Json> j_data{Json{Integer(reinterpret_cast<Integer::Int>(RawPtr(data)))},
                             Json{Boolean{false}}};

    out["data"] = Array(std::move(j_data));
    out["shape"] = Array(std::vector<Json>{Json(Integer(interface.Shape<0>()))});

    if (interface.valid.Data()) {
      CopyColumnMask(interface, columns, kind, c, &mask, &out, stream);
    }
    out["typestr"] = String("<f4");
    out["version"] = Integer(3);
  }
}

template <typename T>
void CopyMetaInfo(Json *p_interface, dh::device_vector<T> *out, hipStream_t stream) {
  auto &j_interface = *p_interface;
  CHECK_EQ(get<Array const>(j_interface).size(), 1);
  auto object = get<Object>(get<Array>(j_interface)[0]);
  ArrayInterface<1> interface(object);
  out->resize(interface.Shape<0>());
  size_t element_size = interface.ElementSize();
  size_t size = element_size * interface.n;
  dh::safe_cuda(
      hipMemcpyAsync(RawPtr(*out), interface.data, size, hipMemcpyDeviceToDevice, stream));
  j_interface[0]["data"][0] = reinterpret_cast<Integer::Int>(RawPtr(*out));
}

template <typename DCont, typename VCont>
struct DataFrame {
  std::vector<DCont> data;
  std::vector<VCont> valid;
  std::vector<Json> interfaces;
};

namespace {
// constant names
struct Symbols {
  static constexpr StringView kLabel{"label"};
  static constexpr StringView kWeight{"weight"};
  static constexpr StringView kBaseMargin{"baseMargin"};
  static constexpr StringView kQid{"qid"};
};
}  // namespace

class JvmIter {
  JNIEnv *jenv_;
  jobject jiter_;
  int jni_status_;
  jobject last_batch_{nullptr};

 public:
  explicit JvmIter(jobject jiter)
      : jiter_{jiter},
        jni_status_{GlobalJvm()->GetEnv(reinterpret_cast<void **>(&jenv_), JNI_VERSION_1_6)} {}

  void CloseJvmBatch() {
    if (last_batch_) {
      jclass batch_class = CheckJvmCall(jenv_->GetObjectClass(last_batch_), jenv_);
      jmethodID closeMethod = CheckJvmCall(jenv_->GetMethodID(batch_class, "close", "()V"), jenv_);
      jenv_->CallVoidMethod(last_batch_, closeMethod);
      last_batch_ = nullptr;
    }
  }

  auto Status() const { return jni_status_; }

  template <typename Fn>
  bool PullIterFromJVM(Fn &&fn) {
    this->CloseJvmBatch();
    jclass iterClass = jenv_->FindClass("java/util/Iterator");

    jmethodID has_next = CheckJvmCall(jenv_->GetMethodID(iterClass, "hasNext", "()Z"), jenv_);
    jmethodID next =
        CheckJvmCall(jenv_->GetMethodID(iterClass, "next", "()Ljava/lang/Object;"), jenv_);

    if (jenv_->CallBooleanMethod(jiter_, has_next)) {
      // batch should be ColumnBatch from jvm
      jobject batch = CheckJvmCall(jenv_->CallObjectMethod(jiter_, next), jenv_);
      jclass batch_class = CheckJvmCall(jenv_->GetObjectClass(batch), jenv_);
      jmethodID toJson =
          CheckJvmCall(jenv_->GetMethodID(batch_class, "toJson", "()Ljava/lang/String;"), jenv_);

      // Json array interface
      auto jaif = static_cast<jstring>(jenv_->CallObjectMethod(batch, toJson));
      CheckJvmCall(jaif, jenv_);
      char const *cjaif = CheckJvmCall(jenv_->GetStringUTFChars(jaif, nullptr), jenv_);

      fn(cjaif);

      jenv_->ReleaseStringUTFChars(jaif, cjaif);

      last_batch_ = batch;
      return true;
    } else {
      return false;
    }
  }
};

class DMatrixProxy {
  DMatrixHandle proxy_;

 public:
  DMatrixProxy() { CHECK_EQ(XGProxyDMatrixCreate(&proxy_), 0); }
  ~DMatrixProxy() { CHECK_EQ(XGDMatrixFree(proxy_), 0); }
  auto GetDMatrixHandle() const { return proxy_; }

  void SetInfo(StringView name, Json jaif) {
    std::string str;
    Json::Dump(jaif, &str);
    CHECK_EQ(XGDMatrixSetInfoFromInterface(proxy_, name.c_str(), str.c_str()), 0);
  }
  void SetData(Json jaif) {
    std::string str;
    Json::Dump(jaif, &str);
    CHECK_EQ(XGProxyDMatrixSetDataCudaColumnar(proxy_, str.c_str()), 0);
  }
};

class DataIteratorProxy {
  DMatrixProxy proxy_;
  JvmIter jiter_;

  template <typename T>
  using Alloc = xgboost::common::cuda_impl::PinnedAllocator<T>;
  template <typename U>
  using HostVector = std::vector<U, Alloc<U>>;

  // This vector is created for staging device data on host to save GPU memory.
  // When space is not of concern, we can stage them on device memory directly.
  std::vector<std::unique_ptr<DataFrame<HostVector<char>, HostVector<std::uint8_t>>>> host_columns_;

  // Staging area for metainfo.
  // TODO(Bobby): label_upper_bound, label_lower_bound.
  std::vector<std::unique_ptr<dh::device_vector<float>>> labels_;
  std::vector<std::unique_ptr<dh::device_vector<float>>> weights_;
  std::vector<std::unique_ptr<dh::device_vector<float>>> base_margins_;
  std::vector<std::unique_ptr<dh::device_vector<int>>> qids_;
  std::vector<Json> label_interfaces_;
  std::vector<Json> weight_interfaces_;
  std::vector<Json> margin_interfaces_;
  std::vector<Json> qid_interfaces_;

  std::size_t it_{0};
  std::size_t n_batches_{0};
  bool initialized_{false};

  // Temp buffer on device, each `dh::device_vector` represents a column
  // from cudf.
  std::vector<dh::device_vector<char>> staging_data_;
  std::vector<dh::device_vector<std::uint8_t>> staging_mask_;

  hipStream_t copy_stream_;

 public:
  explicit DataIteratorProxy(jobject jiter) : jiter_{jiter} {
    this->Reset();
    dh::safe_cuda(hipStreamCreateWithFlags(&copy_stream_, hipStreamNonBlocking));
  }
  ~DataIteratorProxy() { dh::safe_cuda(hipStreamDestroy(copy_stream_)); }

  DMatrixHandle GetDMatrixHandle() const { return proxy_.GetDMatrixHandle(); }

  // Helper function for staging meta info.
  void StageMetaInfo(Json json_interface) {
    CHECK(!IsA<Null>(json_interface));
    auto json_map = get<Object const>(json_interface);
    auto it = json_map.find(Symbols::kLabel);
    if (it == json_map.cend()) {
      LOG(FATAL) << "Must have a label field.";
    }

    Json label = json_interface[Symbols::kLabel.c_str()];
    CHECK(!IsA<Null>(label));
    labels_.emplace_back(std::make_unique<dh::device_vector<float>>());
    CopyMetaInfo(&label, labels_.back().get(), copy_stream_);
    label_interfaces_.emplace_back(label);
    proxy_.SetInfo(Symbols::kLabel, label);

    it = json_map.find(Symbols::kWeight);
    if (it != json_map.cend()) {
      Json weight = json_interface[Symbols::kWeight.c_str()];
      CHECK(!IsA<Null>(weight));
      weights_.emplace_back(new dh::device_vector<float>);
      CopyMetaInfo(&weight, weights_.back().get(), copy_stream_);
      weight_interfaces_.emplace_back(weight);

      proxy_.SetInfo(Symbols::kWeight, weight);
    }

    it = json_map.find(Symbols::kBaseMargin);
    if (it != json_map.cend()) {
      Json base_margin = json_interface[Symbols::kBaseMargin.c_str()];
      base_margins_.emplace_back(new dh::device_vector<float>);
      CopyMetaInfo(&base_margin, base_margins_.back().get(), copy_stream_);
      margin_interfaces_.emplace_back(base_margin);

      proxy_.SetInfo("base_margin", base_margin);
    }

    it = json_map.find(Symbols::kQid);
    if (it != json_map.cend()) {
      Json qid = json_interface[Symbols::kQid.c_str()];
      qids_.emplace_back(new dh::device_vector<int>);
      CopyMetaInfo(&qid, qids_.back().get(), copy_stream_);
      qid_interfaces_.emplace_back(qid);

      proxy_.SetInfo(Symbols::kQid, qid);
    }
  }

  void Reset() {
    it_ = 0;
    this->jiter_.CloseJvmBatch();
  }

  void StageData(std::string interface_str) {
    ++n_batches_;
    // DataFrame
    using T = decltype(host_columns_)::value_type::element_type;
    host_columns_.emplace_back(std::make_unique<T>());

    // Stage the meta info.
    auto json_interface = Json::Load({interface_str.c_str(), interface_str.size()});
    CHECK(!IsA<Null>(json_interface));

    StageMetaInfo(json_interface);

    Json features = json_interface["features"];
    auto json_columns = get<Array const>(features);
    std::vector<ArrayInterface<1>> interfaces;

    // Stage the data
    for (auto &json_col : json_columns) {
      auto column = ArrayInterface<1>(get<Object const>(json_col));
      interfaces.emplace_back(column);
    }
    Json::Dump(features, &interface_str);
    CopyInterface(interfaces, json_columns, hipMemcpyDeviceToHost, &host_columns_.back()->data,
                  &host_columns_.back()->valid, &host_columns_.back()->interfaces, copy_stream_);

    proxy_.SetData(features);
    it_++;
  }

  int NextFirstLoop() {
    try {
      dh::safe_cuda(hipStreamSynchronize(copy_stream_));
      if (this->jiter_.PullIterFromJVM([this](char const *cjaif) { this->StageData(cjaif); })) {
        return 1;
      } else {
        initialized_ = true;
        return 0;
      }
    } catch (dmlc::Error const &e) {
      if (jiter_.Status() == JNI_EDETACHED) {
        GlobalJvm()->DetachCurrentThread();
      }
      LOG(FATAL) << e.what();
    }
    LOG(FATAL) << "Unreachable";
    return 1;
  }

  int NextSecondLoop() {
    std::string str;
    // Meta
    auto const &label = this->label_interfaces_.at(it_);
    proxy_.SetInfo(Symbols::kLabel, label);

    if (n_batches_ == this->weight_interfaces_.size()) {
      auto const &weight = this->weight_interfaces_.at(it_);
      proxy_.SetInfo(Symbols::kWeight, weight);
    }

    if (n_batches_ == this->margin_interfaces_.size()) {
      auto const &base_margin = this->margin_interfaces_.at(it_);
      proxy_.SetInfo("base_margin", base_margin);
    }

    if (n_batches_ == this->qid_interfaces_.size()) {
      auto const &qid = this->qid_interfaces_.at(it_);
      proxy_.SetInfo(Symbols::kQid, qid);
    }

    // Data
    auto const &json_interface = host_columns_.at(it_)->interfaces;

    std::vector<ArrayInterface<1>> in;
    for (auto interface : json_interface) {
      auto column = ArrayInterface<1>(get<Object const>(interface));
      in.emplace_back(column);
    }
    std::vector<Json> out;
    CopyInterface(in, json_interface, hipMemcpyHostToDevice, &staging_data_, &staging_mask_, &out,
                  nullptr);

    Json temp{Array(std::move(out))};
    proxy_.SetData(temp);
    it_++;
    return 1;
  }

  int Next() {
    if (!initialized_) {
      return NextFirstLoop();
    } else {
      if (it_ == n_batches_) {
        return 0;
      }
      return NextSecondLoop();
    }
  };
};

namespace {
void Reset(DataIterHandle self) {
  static_cast<xgboost::jni::DataIteratorProxy *>(self)->Reset();
}

int Next(DataIterHandle self) {
  return static_cast<xgboost::jni::DataIteratorProxy *>(self)->Next();
}

template <typename T>
using Deleter = std::function<void(T *)>;
} // anonymous namespace

XGB_DLL int XGQuantileDMatrixCreateFromCallbackImpl(JNIEnv *jenv, jclass, jobject jdata_iter,
                                                    jlongArray jref, char const *config,
                                                    jlongArray jout) {
  xgboost::jni::DataIteratorProxy proxy(jdata_iter);
  DMatrixHandle result;
  DMatrixHandle ref{nullptr};

  if (jref != nullptr) {
    std::unique_ptr<jlong, Deleter<jlong>> refptr{jenv->GetLongArrayElements(jref, nullptr),
                                                  [&](jlong *ptr) {
                                                    jenv->ReleaseLongArrayElements(jref, ptr, 0);
                                                    jenv->DeleteLocalRef(jref);
                                                  }};
    ref = reinterpret_cast<DMatrixHandle>(refptr.get()[0]);
  }

  auto ret = XGQuantileDMatrixCreateFromCallback(&proxy, proxy.GetDMatrixHandle(), ref, Reset, Next,
                                                 config, &result);
  setHandle(jenv, jout, result);
  return ret;
}
} // namespace xgboost::jni
