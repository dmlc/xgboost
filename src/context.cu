/**
 * Copyright 2022 by XGBoost Contributors
 */
#include "common/cuda_context.cuh"  // HIPContext
#include "xgboost/context.h"

namespace xgboost {
HIPContext const* Context::CUDACtx() const {
  if (!cuctx_) {
    cuctx_.reset(new HIPContext{});
  }
  return cuctx_.get();
}
}  // namespace xgboost
