#include "hip/hip_runtime.h"
/*!
 * Copyright 2020 XGBoost contributors
 */
#include <memory>
#include <type_traits>
#include <algorithm>

#include "../common/hist_util.cuh"
#include "simple_batch_iterator.h"
#include "iterative_device_dmatrix.h"
#include "sparse_page_source.h"
#include "ellpack_page.cuh"
#include "proxy_dmatrix.h"
#include "device_adapter.cuh"

namespace xgboost {
namespace data {

template <typename Fn>
decltype(auto) Dispatch(DMatrixProxy const* proxy, Fn fn) {
  if (proxy->Adapter().type() == typeid(std::shared_ptr<CupyAdapter>)) {
    auto value = dmlc::get<std::shared_ptr<CupyAdapter>>(
        proxy->Adapter())->Value();
    return fn(value);
  } else if (proxy->Adapter().type() == typeid(std::shared_ptr<CudfAdapter>)) {
    auto value = dmlc::get<std::shared_ptr<CudfAdapter>>(
        proxy->Adapter())->Value();
    return fn(value);
  } else {
    LOG(FATAL) << "Unknown type: " << proxy->Adapter().type().name();
    auto value = dmlc::get<std::shared_ptr<CudfAdapter>>(
        proxy->Adapter())->Value();
    return fn(value);
  }
}

void IterativeDeviceDMatrix::Initialize(DataIterHandle iter_handle, float missing, int nthread) {
  // A handle passed to external iterator.
  auto handle = static_cast<std::shared_ptr<DMatrix>*>(proxy_);
  CHECK(handle);
  DMatrixProxy* proxy = static_cast<DMatrixProxy*>(handle->get());
  CHECK(proxy);
  // The external iterator
  auto iter = DataIterProxy<DataIterResetCallback, XGDMatrixCallbackNext>{
    iter_handle, reset_, next_};

  dh::XGBCachingDeviceAllocator<char> alloc;

  auto num_rows = [&]() {
    return Dispatch(proxy, [](auto const &value) { return value.NumRows(); });
  };
  auto num_cols = [&]() {
    return Dispatch(proxy, [](auto const &value) { return value.NumCols(); });
  };

  size_t row_stride = 0;
  size_t nnz = 0;
  // Sketch for all batches.
  iter.Reset();

  std::vector<common::SketchContainer> sketch_containers;
  size_t batches = 0;
  size_t accumulated_rows = 0;
  bst_feature_t cols = 0;
  int32_t device = GenericParameter::kCpuId;
  int32_t current_device_;
  dh::safe_cuda(hipGetDevice(&current_device_));
  auto get_device = [&]() -> int32_t {
    int32_t d = GenericParameter::kCpuId ? current_device_ : device;
    return d;
  };

  while (iter.Next()) {
    device = proxy->DeviceIdx();
    dh::safe_cuda(hipSetDevice(get_device()));
    if (cols == 0) {
      cols = num_cols();
      rabit::Allreduce<rabit::op::Max>(&cols, 1);
    } else {
      CHECK_EQ(cols, num_cols()) << "Inconsistent number of columns.";
    }
    sketch_containers.emplace_back(batch_param_.max_bin, cols, num_rows(), get_device());
    auto* p_sketch = &sketch_containers.back();
    proxy->Info().weights_.SetDevice(get_device());
    Dispatch(proxy, [&](auto const &value) {
        common::AdapterDeviceSketch(value, batch_param_.max_bin,
                                    proxy->Info(), missing, p_sketch);
      });
    auto batch_rows = num_rows();
    accumulated_rows += batch_rows;
    dh::caching_device_vector<size_t> row_counts(batch_rows + 1, 0);
    common::Span<size_t> row_counts_span(row_counts.data().get(),
                                         row_counts.size());
    row_stride = std::max(row_stride, Dispatch(proxy, [=](auto const &value) {
          return GetRowCounts(value, row_counts_span,
                              get_device(), missing);
        }));
    nnz += thrust::reduce(thrust::cuda::par(alloc), row_counts.begin(),
                          row_counts.end());
    batches++;
  }
  iter.Reset();
  dh::safe_cuda(hipSetDevice(get_device()));
  common::SketchContainer final_sketch(batch_param_.max_bin, cols, accumulated_rows, get_device());
  for (auto const& sketch : sketch_containers) {
    final_sketch.Merge(sketch.ColumnsPtr(), sketch.Data());
    final_sketch.FixError();
  }
  sketch_containers.clear();
  sketch_containers.shrink_to_fit();

  common::HistogramCuts cuts;
  final_sketch.MakeCuts(&cuts);

  this->info_.num_col_ = cols;
  this->info_.num_row_ = accumulated_rows;
  this->info_.num_nonzero_ = nnz;

  auto init_page = [this, &proxy, &cuts, row_stride, accumulated_rows,
                    get_device]() {
    if (!page_) {
      // Should be put inside the while loop to protect against empty batch.  In
      // that case device id is invalid.
      page_.reset(new EllpackPage);
      *(page_->Impl()) = EllpackPageImpl(get_device(), cuts, this->IsDense(),
                                         row_stride, accumulated_rows);
    }
  };

  // Construct the final ellpack page.
  size_t offset = 0;
  iter.Reset();
  size_t n_batches_for_verification = 0;
  while (iter.Next()) {
    init_page();
    dh::safe_cuda(hipSetDevice(get_device()));
    auto rows = num_rows();
    dh::caching_device_vector<size_t> row_counts(rows + 1, 0);
    common::Span<size_t> row_counts_span(row_counts.data().get(),
                                         row_counts.size());
    Dispatch(proxy, [=](auto const& value) {
        return GetRowCounts(value, row_counts_span, get_device(), missing);
      });
    auto is_dense = this->IsDense();
    auto new_impl = Dispatch(proxy, [&](auto const &value) {
        return EllpackPageImpl(value, missing, get_device(), is_dense, nthread,
                               row_counts_span, row_stride, rows, cols, cuts);
    });
    size_t num_elements = page_->Impl()->Copy(get_device(), &new_impl, offset);
    offset += num_elements;

    proxy->Info().num_row_ = num_rows();
    proxy->Info().num_col_ = cols;
    if (batches != 1) {
      this->info_.Extend(std::move(proxy->Info()), false);
    }
    n_batches_for_verification++;
  }
  CHECK_EQ(batches, n_batches_for_verification)
      << "Different number of batches returned between 2 iterations";

  if (batches == 1) {
    this->info_ = std::move(proxy->Info());
    CHECK_EQ(proxy->Info().labels_.Size(), 0);
  }

  iter.Reset();
  // Synchronise worker columns
  rabit::Allreduce<rabit::op::Max>(&info_.num_col_, 1);
}

BatchSet<EllpackPage> IterativeDeviceDMatrix::GetEllpackBatches(const BatchParam& param) {
  CHECK(page_);
  auto begin_iter =
      BatchIterator<EllpackPage>(new SimpleBatchIteratorImpl<EllpackPage>(page_.get()));
  return BatchSet<EllpackPage>(begin_iter);
}
}  // namespace data
}  // namespace xgboost
