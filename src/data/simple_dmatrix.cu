#include "hip/hip_runtime.h"
/*!
 * Copyright 2019 by Contributors
 * \file simple_dmatrix.cu
 */
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <xgboost/data.h>
#include "../common/random.h"
#include "./simple_dmatrix.h"
#include "device_adapter.cuh"

namespace xgboost {
namespace data {


template <typename AdapterBatchT>
void CountRowOffsets(const AdapterBatchT& batch, common::Span<bst_row_t> offset,
                     int device_idx, float missing) {
  IsValidFunctor is_valid(missing);
  // Count elements per row
  dh::LaunchN(device_idx, batch.Size(), [=] __device__(size_t idx) {
    auto element = batch.GetElement(idx);
    if (is_valid(element)) {
      atomicAdd(reinterpret_cast<unsigned long long*>(  // NOLINT
                    &offset[element.row_idx]),
                static_cast<unsigned long long>(1));  // NOLINT
    }
  });

  dh::XGBCachingDeviceAllocator<char> alloc;
  thrust::exclusive_scan(thrust::cuda::par(alloc),
      thrust::device_pointer_cast(offset.data()),
      thrust::device_pointer_cast(offset.data() + offset.size()),
      thrust::device_pointer_cast(offset.data()));
}

// Here the data is already correctly ordered and simply needs to be compacted
// to remove missing data
template <typename AdapterT>
void CopyDataToDMatrix(AdapterT* adapter, common::Span<Entry> data,
                       int device_idx, float missing,
                       common::Span<size_t> row_ptr) {
  auto& batch = adapter->Value();
  auto transform_f = [=] __device__(size_t idx) {
    const auto& e = batch.GetElement(idx);
    return Entry(e.column_idx, e.value);
  };  // NOLINT
  auto counting = thrust::make_counting_iterator(0llu);
  thrust::transform_iterator<decltype(transform_f), decltype(counting), Entry>
      transform_iter(counting, transform_f);
  dh::XGBCachingDeviceAllocator<char> alloc;
  thrust::copy_if(
      thrust::cuda::par(alloc), transform_iter, transform_iter + batch.Size(),
      thrust::device_pointer_cast(data.data()), IsValidFunctor(missing));
}

// Does not currently support metainfo as no on-device data source contains this
// Current implementation assumes a single batch. More batches can
// be supported in future. Does not currently support inferring row/column size
template <typename AdapterT>
SimpleDMatrix::SimpleDMatrix(AdapterT* adapter, float missing, int nthread) {
  dh::safe_cuda(hipSetDevice(adapter->DeviceIdx()));
  CHECK(adapter->NumRows() != kAdapterUnknownSize);
  CHECK(adapter->NumColumns() != kAdapterUnknownSize);

  adapter->BeforeFirst();
  adapter->Next();
  auto& batch = adapter->Value();
  sparse_page_.offset.SetDevice(adapter->DeviceIdx());
  sparse_page_.data.SetDevice(adapter->DeviceIdx());

  // Enforce single batch
  CHECK(!adapter->Next());
  sparse_page_.offset.Resize(adapter->NumRows() + 1);
  auto s_offset = sparse_page_.offset.DeviceSpan();
  CountRowOffsets(batch, s_offset, adapter->DeviceIdx(), missing);
  info_.num_nonzero_ = sparse_page_.offset.HostVector().back();
  sparse_page_.data.Resize(info_.num_nonzero_);
  CopyDataToDMatrix(adapter, sparse_page_.data.DeviceSpan(),
                    adapter->DeviceIdx(), missing, s_offset);

  info_.num_col_ = adapter->NumColumns();
  info_.num_row_ = adapter->NumRows();
  // Synchronise worker columns
  rabit::Allreduce<rabit::op::Max>(&info_.num_col_, 1);
}

template SimpleDMatrix::SimpleDMatrix(CudfAdapter* adapter, float missing,
                                      int nthread);
template SimpleDMatrix::SimpleDMatrix(CupyAdapter* adapter, float missing,
                                      int nthread);
}  // namespace data
}  // namespace xgboost
