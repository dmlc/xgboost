/**
 * Copyright 2019-2025, XGBoost Contributors
 */

#include <cstdint>  // for int32_t, int8_t
#include <memory>   // for make_shared

#include "../common/cuda_rt_utils.h"  // for CurrentDevice
#include "cat_container.h"            // for CatContainer
#include "device_adapter.cuh"
#include "simple_dmatrix.cuh"
#include "simple_dmatrix.h"
#include "xgboost/context.h"  // for Context
#include "xgboost/data.h"

namespace xgboost::data {
// Does not currently support metainfo as no on-device data source contains this
// Current implementation assumes a single batch. More batches can
// be supported in future. Does not currently support inferring row/column size
template <typename AdapterT>
SimpleDMatrix::SimpleDMatrix(AdapterT* adapter, float missing, std::int32_t nthread,
                             DataSplitMode data_split_mode) {
  CHECK(data_split_mode != DataSplitMode::kCol)
      << "Column-wise data split is currently not supported on the GPU.";
  auto device = (!adapter->Device().IsCUDA() || adapter->NumRows() == 0)
                    ? DeviceOrd::CUDA(curt::CurrentDevice())
                    : adapter->Device();
  CHECK(device.IsCUDA());
  dh::safe_cuda(hipSetDevice(device.ordinal));

  Context ctx;
  ctx.Init(Args{{"nthread", std::to_string(nthread)}, {"device", device.Name()}});

  CHECK(adapter->NumRows() != kAdapterUnknownSize);
  CHECK(adapter->NumColumns() != kAdapterUnknownSize);

  adapter->BeforeFirst();
  adapter->Next();

  // Enforce single batch
  CHECK(!adapter->Next());

  info_.num_nonzero_ =
      CopyToSparsePage(&ctx, adapter->Value(), device, missing, sparse_page_.get());
  info_.num_col_ = adapter->NumColumns();
  info_.num_row_ = adapter->NumRows();

  if constexpr (std::is_same_v<AdapterT, CudfAdapter>) {
    if (adapter->HasCategorical()) {
      info_.Cats(std::make_shared<CatContainer>(adapter->Device(), adapter->Cats()));
    }
  }
  this->info_.SynchronizeNumberOfColumns(&ctx, data_split_mode);

  this->fmat_ctx_ = ctx;
}

template SimpleDMatrix::SimpleDMatrix(CudfAdapter* adapter, float missing, std::int32_t nthread,
                                      DataSplitMode data_split_mode);
template SimpleDMatrix::SimpleDMatrix(CupyAdapter* adapter, float missing, std::int32_t nthread,
                                      DataSplitMode data_split_mode);
}  // namespace xgboost::data
