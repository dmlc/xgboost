/*!
 * Copyright 2019-2021 by XGBoost Contributors
 * \file simple_dmatrix.cu
 */
#include <thrust/copy.h>
#include <xgboost/data.h>
#include "simple_dmatrix.cuh"
#include "simple_dmatrix.h"
#include "device_adapter.cuh"

namespace xgboost {
namespace data {

// Does not currently support metainfo as no on-device data source contains this
// Current implementation assumes a single batch. More batches can
// be supported in future. Does not currently support inferring row/column size
template <typename AdapterT>
SimpleDMatrix::SimpleDMatrix(AdapterT* adapter, float missing, int32_t /*nthread*/) {
  auto device = (adapter->DeviceIdx() < 0 || adapter->NumRows() == 0) ? dh::CurrentDevice()
                                                                      : adapter->DeviceIdx();
  CHECK_GE(device, 0);
  dh::safe_cuda(hipSetDevice(device));

  CHECK(adapter->NumRows() != kAdapterUnknownSize);
  CHECK(adapter->NumColumns() != kAdapterUnknownSize);

  adapter->BeforeFirst();
  adapter->Next();

  // Enforce single batch
  CHECK(!adapter->Next());

  info_.num_nonzero_ =
      CopyToSparsePage(adapter->Value(), device, missing, sparse_page_.get());
  info_.num_col_ = adapter->NumColumns();
  info_.num_row_ = adapter->NumRows();
  // Synchronise worker columns
  rabit::Allreduce<rabit::op::Max>(&info_.num_col_, 1);
}

template SimpleDMatrix::SimpleDMatrix(CudfAdapter* adapter, float missing,
                                      int nthread);
template SimpleDMatrix::SimpleDMatrix(CupyAdapter* adapter, float missing,
                                      int nthread);
}  // namespace data
}  // namespace xgboost
