#include "hip/hip_runtime.h"
/*!
 * Copyright 2019 by Contributors
 * \file simple_dmatrix.cu
 */
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <xgboost/data.h>
#include "../common/random.h"
#include "./simple_dmatrix.h"
#include "device_adapter.cuh"

namespace xgboost {
namespace data {


template <typename AdapterBatchT>
void CountRowOffsets(const AdapterBatchT& batch, common::Span<bst_row_t> offset,
                     int device_idx, float missing) {
  IsValidFunctor is_valid(missing);
  // Count elements per row
  dh::LaunchN(device_idx, batch.Size(), [=] __device__(size_t idx) {
    auto element = batch.GetElement(idx);
    if (is_valid(element)) {
      atomicAdd(reinterpret_cast<unsigned long long*>(  // NOLINT
                    &offset[element.row_idx]),
                static_cast<unsigned long long>(1));  // NOLINT
    }
  });

  dh::XGBCachingDeviceAllocator<char> alloc;
  thrust::exclusive_scan(thrust::cuda::par(alloc),
      thrust::device_pointer_cast(offset.data()),
      thrust::device_pointer_cast(offset.data() + offset.size()),
      thrust::device_pointer_cast(offset.data()));
}

template <typename AdapterBatchT>
struct COOToEntryOp {
  AdapterBatchT batch;
  __device__ Entry operator()(size_t idx) {
    const auto& e = batch.GetElement(idx);
    return Entry(e.column_idx, e.value);
  }
};

// Here the data is already correctly ordered and simply needs to be compacted
// to remove missing data
template <typename AdapterT>
void CopyDataToDMatrix(AdapterT* adapter, common::Span<Entry> data,
                       float missing) {
  auto batch = adapter->Value();
  auto counting = thrust::make_counting_iterator(0llu);
  dh::XGBCachingDeviceAllocator<char> alloc;
  COOToEntryOp<decltype(batch)> transform_op{batch};
  thrust::transform_iterator<decltype(transform_op), decltype(counting)>
      transform_iter(counting, transform_op);
  // We loop over batches because thrust::copy_if cant deal with sizes > 2^31
  // See thrust issue #1302
  size_t max_copy_size = std::numeric_limits<int>::max() / 2;
  auto begin_output = thrust::device_pointer_cast(data.data());
  for (size_t offset = 0; offset < batch.Size(); offset += max_copy_size) {
    auto begin_input = transform_iter + offset;
    auto end_input =
        transform_iter + std::min(offset + max_copy_size, batch.Size());
    begin_output =
        thrust::copy_if(thrust::cuda::par(alloc), begin_input, end_input,
                        begin_output, IsValidFunctor(missing));
  }
}

// Does not currently support metainfo as no on-device data source contains this
// Current implementation assumes a single batch. More batches can
// be supported in future. Does not currently support inferring row/column size
template <typename AdapterT>
SimpleDMatrix::SimpleDMatrix(AdapterT* adapter, float missing, int nthread) {
  dh::safe_cuda(hipSetDevice(adapter->DeviceIdx()));
  CHECK(adapter->NumRows() != kAdapterUnknownSize);
  CHECK(adapter->NumColumns() != kAdapterUnknownSize);

  adapter->BeforeFirst();
  adapter->Next();
  auto& batch = adapter->Value();
  sparse_page_.offset.SetDevice(adapter->DeviceIdx());
  sparse_page_.data.SetDevice(adapter->DeviceIdx());

  // Enforce single batch
  CHECK(!adapter->Next());
  sparse_page_.offset.Resize(adapter->NumRows() + 1);
  auto s_offset = sparse_page_.offset.DeviceSpan();
  CountRowOffsets(batch, s_offset, adapter->DeviceIdx(), missing);
  info_.num_nonzero_ = sparse_page_.offset.HostVector().back();
  sparse_page_.data.Resize(info_.num_nonzero_);
  CopyDataToDMatrix(adapter, sparse_page_.data.DeviceSpan(), missing);

  info_.num_col_ = adapter->NumColumns();
  info_.num_row_ = adapter->NumRows();
  // Synchronise worker columns
  rabit::Allreduce<rabit::op::Max>(&info_.num_col_, 1);
}

template SimpleDMatrix::SimpleDMatrix(CudfAdapter* adapter, float missing,
                                      int nthread);
template SimpleDMatrix::SimpleDMatrix(CupyAdapter* adapter, float missing,
                                      int nthread);
}  // namespace data
}  // namespace xgboost
