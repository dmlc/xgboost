#include "hip/hip_runtime.h"
/**
 * Copyright 2019-2025, XGBoost Contributors
 */
#include "../common/cuda_rt_utils.h"  // for SetDevice, CurrentDevice
#include "columnar.h"                 // for GetRefCats, GetArrowDictionary
#include "device_adapter.cuh"

namespace xgboost::data {
namespace {
auto GetRefCats(Context const* ctx, Json handle,
                std::vector<enc::DeviceCatIndexView>* p_h_ref_cats) {
  auto& h_ref_cats = *p_h_ref_cats;
  auto cats = reinterpret_cast<CatContainer const*>(get<Integer const>(handle));
  CHECK(cats);
  auto d_cats = cats->DeviceView(ctx);
  // FIXME(jiamingy): Remove this along with the host copy in the cat container once
  // cuDF can return device-only data.
  h_ref_cats.resize(d_cats.columns.size());
  thrust::copy(dh::tcbegin(d_cats.columns), dh::tcend(d_cats.columns), h_ref_cats.begin());
  d_cats.columns = common::Span{h_ref_cats};
  return d_cats;
}
}  // anonymous namespace

CudfAdapter::CudfAdapter(StringView cuda_arrinf) {
  Json jdf = Json::Load(cuda_arrinf);

  if (IsA<Object>(jdf)) {
    // Has reference categories.
    auto ctx = Context{}.MakeCUDA(curt::CurrentDevice());
    this->ref_cats_ = GetRefCats(&ctx, jdf["ref_categories"], &this->h_ref_cats_);
    jdf = jdf["columns"];
  }

  std::vector<Json> const& jcolumns = get<Array>(jdf);
  std::size_t n_columns = jcolumns.size();
  CHECK_GT(n_columns, 0) << "The number of columns must not equal to 0.";

  std::vector<ArrayInterface<1>> columns;
  std::vector<std::int32_t> cat_segments{0};
  std::int32_t device = -1;
  for (auto const& jcol : jcolumns) {
    std::int32_t n_cats{0};
    if (IsA<Array>(jcol)) {
      // This is a dictionary type (categorical values).
      auto const& first = get<Object const>(jcol[0]);
      if (first.find("offsets") == first.cend()) {
        // numeric index
        if (device == -1) {
          auto const& first = get<Object const>(jcol[0]);
          auto names = ArrayInterface<1>{first};
          device = dh::CudaGetPointerDevice(names.data);
        }
        n_cats = GetArrowNumericIndex(DeviceOrd::CUDA(device), jcol, &cats_, &columns, &n_bytes_,
                                      &num_rows_);
      } else {
        // string index
        n_cats = GetArrowDictionary(jcol, &cats_, &columns, &n_bytes_, &num_rows_);
      }
    } else {
      // Numeric values
      auto col = ArrayInterface<1>(get<Object const>(jcol));
      columns.push_back(col);
      this->cats_.emplace_back();
      this->num_rows_ = std::max(num_rows_, col.Shape<0>());
      CHECK_EQ(num_rows_, col.Shape<0>()) << "All columns should have the same number of rows.";
      n_bytes_ += col.ElementSize() * col.Shape<0>();
    }
    cat_segments.emplace_back(n_cats);
    if (device == -1) {
      device = dh::CudaGetPointerDevice(columns.back().data);
    }
    CHECK_EQ(device, dh::CudaGetPointerDevice(columns.back().data))
        << "All columns should use the same device.";
  }
  // Categories
  std::partial_sum(cat_segments.cbegin(), cat_segments.cend(), cat_segments.begin());
  this->n_total_cats_ = cat_segments.back();
  this->cat_segments_ = std::move(cat_segments);
  this->d_cats_ = this->cats_;  // thrust copy

  CHECK(!columns.empty());
  if (device < 0) {
    // Empty dataset
    CHECK_EQ(columns.front().Shape<0>(), 0);
    device_ = DeviceOrd::CUDA(curt::CurrentDevice());
  } else {
    device_ = DeviceOrd::CUDA(device);
  }
  CHECK(device_.IsCUDA());
  curt::SetDevice(device_.ordinal);

  this->columns_ = columns;
  batch_ = CudfAdapterBatch(dh::ToSpan(columns_), NoOpAccessor{}, num_rows_);

  if (!this->ref_cats_.Empty()) {
    CHECK_EQ(this->ref_cats_.Size(), this->columns_.size())
        << "Invalid reference categories, different number of columns";
  }
}
}  // namespace xgboost::data
