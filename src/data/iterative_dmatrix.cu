#include "hip/hip_runtime.h"
/*!
 * Copyright 2020-2022 XGBoost contributors
 */
#include <algorithm>
#include <memory>
#include <type_traits>

#include "../common/hist_util.cuh"
#include "device_adapter.cuh"
#include "ellpack_page.cuh"
#include "iterative_dmatrix.h"
#include "proxy_dmatrix.cuh"
#include "proxy_dmatrix.h"
#include "simple_batch_iterator.h"
#include "sparse_page_source.h"

namespace xgboost {
namespace data {
void IterativeDMatrix::InitFromCUDA(DataIterHandle iter_handle, float missing,
                                    std::shared_ptr<DMatrix> ref) {
  // A handle passed to external iterator.
  DMatrixProxy* proxy = MakeProxy(proxy_);
  CHECK(proxy);

  // The external iterator
  auto iter =
      DataIterProxy<DataIterResetCallback, XGDMatrixCallbackNext>{iter_handle, reset_, next_};

  dh::XGBCachingDeviceAllocator<char> alloc;

  auto num_rows = [&]() {
    return Dispatch(proxy, [](auto const& value) { return value.NumRows(); });
  };
  auto num_cols = [&]() {
    return Dispatch(proxy, [](auto const& value) { return value.NumCols(); });
  };

  size_t row_stride = 0;
  size_t nnz = 0;
  // Sketch for all batches.
  std::vector<common::SketchContainer> sketch_containers;
  size_t batches = 0;
  size_t accumulated_rows = 0;
  bst_feature_t cols = 0;

  int32_t current_device;
  dh::safe_cuda(hipGetDevice(&current_device));
  auto get_device = [&]() -> int32_t {
    int32_t d = (ctx_.gpu_id == Context::kCpuId) ? current_device : ctx_.gpu_id;
    CHECK_NE(d, Context::kCpuId);
    return d;
  };

  /**
   * Generate quantiles
   */
  common::HistogramCuts cuts;
  do {
    // We use do while here as the first batch is fetched in ctor
    ctx_.gpu_id = proxy->DeviceIdx();
    CHECK_LT(ctx_.gpu_id, common::AllVisibleGPUs());
    dh::safe_cuda(hipSetDevice(get_device()));
    if (cols == 0) {
      cols = num_cols();
      rabit::Allreduce<rabit::op::Max>(&cols, 1);
      this->info_.num_col_ = cols;
    } else {
      CHECK_EQ(cols, num_cols()) << "Inconsistent number of columns.";
    }
    if (!ref) {
      sketch_containers.emplace_back(proxy->Info().feature_types, batch_param_.max_bin, cols,
                                     num_rows(), get_device());
      auto* p_sketch = &sketch_containers.back();
      proxy->Info().weights_.SetDevice(get_device());
      Dispatch(proxy, [&](auto const& value) {
        common::AdapterDeviceSketch(value, batch_param_.max_bin, proxy->Info(), missing, p_sketch);
      });
    }
    auto batch_rows = num_rows();
    accumulated_rows += batch_rows;
    dh::caching_device_vector<size_t> row_counts(batch_rows + 1, 0);
    common::Span<size_t> row_counts_span(row_counts.data().get(), row_counts.size());
    row_stride = std::max(row_stride, Dispatch(proxy, [=](auto const& value) {
                            return GetRowCounts(value, row_counts_span, get_device(), missing);
                          }));
    nnz += thrust::reduce(thrust::cuda::par(alloc), row_counts.begin(), row_counts.end());
    batches++;
  } while (iter.Next());
  iter.Reset();

  dh::safe_cuda(hipSetDevice(get_device()));
  if (!ref) {
    HostDeviceVector<FeatureType> ft;
    common::SketchContainer final_sketch(
        sketch_containers.empty() ? ft : sketch_containers.front().FeatureTypes(),
        batch_param_.max_bin, cols, accumulated_rows, get_device());
    for (auto const& sketch : sketch_containers) {
      final_sketch.Merge(sketch.ColumnsPtr(), sketch.Data());
      final_sketch.FixError();
    }
    sketch_containers.clear();
    sketch_containers.shrink_to_fit();

    final_sketch.MakeCuts(&cuts);
  } else {
    GetCutsFromRef(ref, Info().num_col_, batch_param_, &cuts);
  }

  this->info_.num_row_ = accumulated_rows;
  this->info_.num_nonzero_ = nnz;

  auto init_page = [this, &proxy, &cuts, row_stride, accumulated_rows, get_device]() {
    if (!ellpack_) {
      // Should be put inside the while loop to protect against empty batch.  In
      // that case device id is invalid.
      ellpack_.reset(new EllpackPage);
      *(ellpack_->Impl()) =
          EllpackPageImpl(get_device(), cuts, this->IsDense(), row_stride, accumulated_rows);
    }
  };

  /**
   * Generate gradient index.
   */
  size_t offset = 0;
  iter.Reset();
  size_t n_batches_for_verification = 0;
  while (iter.Next()) {
    init_page();
    dh::safe_cuda(hipSetDevice(get_device()));
    auto rows = num_rows();
    dh::caching_device_vector<size_t> row_counts(rows + 1, 0);
    common::Span<size_t> row_counts_span(row_counts.data().get(), row_counts.size());
    Dispatch(proxy, [=](auto const& value) {
      return GetRowCounts(value, row_counts_span, get_device(), missing);
    });
    auto is_dense = this->IsDense();

    proxy->Info().feature_types.SetDevice(get_device());
    auto d_feature_types = proxy->Info().feature_types.ConstDeviceSpan();
    auto new_impl = Dispatch(proxy, [&](auto const& value) {
      return EllpackPageImpl(value, missing, get_device(), is_dense, row_counts_span,
                             d_feature_types, row_stride, rows, cuts);
    });
    size_t num_elements = ellpack_->Impl()->Copy(get_device(), &new_impl, offset);
    offset += num_elements;

    proxy->Info().num_row_ = num_rows();
    proxy->Info().num_col_ = cols;
    if (batches != 1) {
      this->info_.Extend(std::move(proxy->Info()), false, true);
    }
    n_batches_for_verification++;
  }
  CHECK_EQ(batches, n_batches_for_verification)
      << "Different number of batches returned between 2 iterations";

  if (batches == 1) {
    this->info_ = std::move(proxy->Info());
    this->info_.num_nonzero_ = nnz;
    CHECK_EQ(proxy->Info().labels.Size(), 0);
  }

  iter.Reset();
  // Synchronise worker columns
  rabit::Allreduce<rabit::op::Max>(&info_.num_col_, 1);
}

BatchSet<EllpackPage> IterativeDMatrix::GetEllpackBatches(BatchParam const& param) {
  CheckParam(param);
  if (!ellpack_ && !ghist_) {
    LOG(FATAL) << "`QuantileDMatrix` not initialized.";
  }
  if (!ellpack_ && ghist_) {
    ellpack_.reset(new EllpackPage());
    // Evaluation QuantileDMatrix initialized from CPU data might not have the correct GPU
    // ID.
    if (this->ctx_.IsCPU()) {
      this->ctx_.gpu_id = param.gpu_id;
    }
    if (this->ctx_.IsCPU()) {
      this->ctx_.gpu_id = dh::CurrentDevice();
    }
    this->Info().feature_types.SetDevice(this->ctx_.gpu_id);
    *ellpack_->Impl() =
        EllpackPageImpl(&ctx_, *this->ghist_, this->Info().feature_types.ConstDeviceSpan());
  }
  CHECK(ellpack_);
  auto begin_iter = BatchIterator<EllpackPage>(new SimpleBatchIteratorImpl<EllpackPage>(ellpack_));
  return BatchSet<EllpackPage>(begin_iter);
}

void GetCutsFromEllpack(EllpackPage const& page, common::HistogramCuts* cuts) {
  *cuts = page.Impl()->Cuts();
}
}  // namespace data
}  // namespace xgboost
