#include "hip/hip_runtime.h"
/*!
 * Copyright 2019 XGBoost contributors
 */

#include <xgboost/data.h>

#include "./ellpack_page.cuh"
#include "../common/hist_util.h"
#include "../common/random.h"

namespace xgboost {

EllpackPage::EllpackPage() : impl_{new EllpackPageImpl()} {}

EllpackPage::EllpackPage(DMatrix* dmat, const BatchParam& param)
    : impl_{new EllpackPageImpl(dmat, param)} {}

EllpackPage::~EllpackPage() = default;

size_t EllpackPage::Size() const {
  return impl_->Size();
}

void EllpackPage::SetBaseRowId(size_t row_id) {
  impl_->SetBaseRowId(row_id);
}

// Bin each input data entry, store the bin indices in compressed form.
__global__ void CompressBinEllpackKernel(
    common::CompressedBufferWriter wr,
    common::CompressedByteT* __restrict__ buffer,  // gidx_buffer
    const size_t* __restrict__ row_ptrs,           // row offset of input data
    const Entry* __restrict__ entries,      // One batch of input data
    const float* __restrict__ cuts,         // HistogramCuts::cut_values_
    const uint32_t* __restrict__ cut_rows,  // HistogramCuts::cut_ptrs_
    size_t base_row,                        // batch_row_begin
    size_t n_rows,
    size_t row_stride,
    unsigned int null_gidx_value) {
  size_t irow = threadIdx.x + blockIdx.x * blockDim.x;
  int ifeature = threadIdx.y + blockIdx.y * blockDim.y;
  if (irow >= n_rows || ifeature >= row_stride) {
    return;
  }
  int row_length = static_cast<int>(row_ptrs[irow + 1] - row_ptrs[irow]);
  unsigned int bin = null_gidx_value;
  if (ifeature < row_length) {
    Entry entry = entries[row_ptrs[irow] - row_ptrs[0] + ifeature];
    int feature = entry.index;
    float fvalue = entry.fvalue;
    // {feature_cuts, ncuts} forms the array of cuts of `feature'.
    const float* feature_cuts = &cuts[cut_rows[feature]];
    int ncuts = cut_rows[feature + 1] - cut_rows[feature];
    // Assigning the bin in current entry.
    // S.t.: fvalue < feature_cuts[bin]
    bin = dh::UpperBound(feature_cuts, ncuts, fvalue);
    if (bin >= ncuts) {
      bin = ncuts - 1;
    }
    // Add the number of bins in previous features.
    bin += cut_rows[feature];
  }
  // Write to gidx buffer.
  wr.AtomicWriteSymbol(buffer, bin, (irow + base_row) * row_stride + ifeature);
}

// Construct an ELLPACK matrix with the given number of empty rows.
EllpackPageImpl::EllpackPageImpl(int device, EllpackInfo info, size_t n_rows) {
  monitor_.Init("ellpack_page");
  dh::safe_cuda(hipSetDevice(device));

  matrix.info = info;
  matrix.base_rowid = 0;
  matrix.n_rows = n_rows;

  monitor_.StartCuda("InitCompressedData");
  InitCompressedData(device, n_rows);
  monitor_.StopCuda("InitCompressedData");
}

// Construct an ELLPACK matrix in memory.
EllpackPageImpl::EllpackPageImpl(DMatrix* dmat, const BatchParam& param) {
  monitor_.Init("ellpack_page");
  dh::safe_cuda(hipSetDevice(param.gpu_id));

  matrix.n_rows = dmat->Info().num_row_;

  monitor_.StartCuda("Quantiles");
  // Create the quantile sketches for the dmatrix and initialize HistogramCuts.
  common::HistogramCuts hmat;
  size_t row_stride =
      common::DeviceSketch(param.gpu_id, param.max_bin, param.gpu_batch_nrows, dmat, &hmat);
  monitor_.StopCuda("Quantiles");

  monitor_.StartCuda("InitEllpackInfo");
  InitInfo(param.gpu_id, dmat->IsDense(), row_stride, hmat);
  monitor_.StopCuda("InitEllpackInfo");

  monitor_.StartCuda("InitCompressedData");
  InitCompressedData(param.gpu_id, dmat->Info().num_row_);
  monitor_.StopCuda("InitCompressedData");

  monitor_.StartCuda("BinningCompression");
  DeviceHistogramBuilderState hist_builder_row_state(dmat->Info().num_row_);
  for (const auto& batch : dmat->GetBatches<SparsePage>()) {
    hist_builder_row_state.BeginBatch(batch);
    CreateHistIndices(param.gpu_id, batch, hist_builder_row_state.GetRowStateOnDevice());
    hist_builder_row_state.EndBatch();
  }
  monitor_.StopCuda("BinningCompression");
}

// A functor that copies the data from one EllpackPage to another.
struct CopyPage {
  common::CompressedBufferWriter cbw;
  common::CompressedByteT* dst_data_d;
  common::CompressedIterator<uint32_t> src_iterator_d;
  // The number of elements to skip.
  size_t offset;

  CopyPage(EllpackPageImpl* dst, EllpackPageImpl* src, size_t offset)
      : cbw{dst->matrix.info.NumSymbols()},
        dst_data_d{dst->gidx_buffer.data()},
        src_iterator_d{src->gidx_buffer.data(), src->matrix.info.NumSymbols()},
        offset(offset) {}

  __device__ void operator()(size_t element_id) {
    cbw.AtomicWriteSymbol(dst_data_d, src_iterator_d[element_id], element_id + offset);
  }
};

// Copy the data from the given EllpackPage to the current page.
size_t EllpackPageImpl::Copy(int device, EllpackPageImpl* page, size_t offset) {
  monitor_.StartCuda("Copy");
  size_t num_elements = page->matrix.n_rows * page->matrix.info.row_stride;
  CHECK_EQ(matrix.info.row_stride, page->matrix.info.row_stride);
  CHECK_EQ(matrix.info.NumSymbols(), page->matrix.info.NumSymbols());
  CHECK_GE(matrix.n_rows * matrix.info.row_stride, offset + num_elements);
  dh::LaunchN(device, num_elements, CopyPage(this, page, offset));
  monitor_.StopCuda("Copy");
  return num_elements;
}

// A functor that compacts the rows from one EllpackPage into another.
struct CompactPage {
  common::CompressedBufferWriter cbw;
  common::CompressedByteT* dst_data_d;
  common::CompressedIterator<uint32_t> src_iterator_d;
  /*! \brief An array that maps the rows from the full DMatrix to the compacted page.
   *
   * The total size is the number of rows in the original, uncompacted DMatrix. Elements are the
   * row ids in the compacted page. Rows not needed are set to SIZE_MAX.
   *
   * An example compacting 16 rows to 8 rows:
   * [SIZE_MAX, 0, 1, SIZE_MAX, SIZE_MAX, 2, SIZE_MAX, 3, 4, 5, SIZE_MAX, 6, SIZE_MAX, 7, SIZE_MAX,
   * SIZE_MAX]
   */
  common::Span<size_t> row_indexes;
  size_t base_rowid;
  size_t row_stride;

  CompactPage(EllpackPageImpl* dst, EllpackPageImpl* src, common::Span<size_t> row_indexes)
      : cbw{dst->matrix.info.NumSymbols()},
        dst_data_d{dst->gidx_buffer.data()},
        src_iterator_d{src->gidx_buffer.data(), src->matrix.info.NumSymbols()},
        row_indexes(row_indexes),
        base_rowid{src->matrix.base_rowid},
        row_stride{src->matrix.info.row_stride} {}

  __device__ void operator()(size_t row_id) {
    size_t src_row = base_rowid + row_id;
    size_t dst_row = row_indexes[src_row];
    if (dst_row == SIZE_MAX) return;
    size_t dst_offset = dst_row * row_stride;
    size_t src_offset = row_id * row_stride;
    for (size_t j = 0; j < row_stride; j++) {
      cbw.AtomicWriteSymbol(dst_data_d, src_iterator_d[src_offset + j], dst_offset + j);
    }
  }
};

// Compacts the data from the given EllpackPage into the current page.
void EllpackPageImpl::Compact(int device, EllpackPageImpl* page, common::Span<size_t> row_indexes) {
  monitor_.StartCuda("Compact");
  CHECK_EQ(matrix.info.row_stride, page->matrix.info.row_stride);
  CHECK_EQ(matrix.info.NumSymbols(), page->matrix.info.NumSymbols());
  CHECK_LE(page->matrix.base_rowid + page->matrix.n_rows, row_indexes.size());
  dh::LaunchN(device, page->matrix.n_rows, CompactPage(this, page, row_indexes));
  monitor_.StopCuda("Compact");
}

// Construct an EllpackInfo based on histogram cuts of features.
EllpackInfo::EllpackInfo(int device,
                         bool is_dense,
                         size_t row_stride,
                         const common::HistogramCuts& hmat,
                         dh::BulkAllocator* ba)
    : is_dense(is_dense), row_stride(row_stride), n_bins(hmat.Ptrs().back()) {

  ba->Allocate(device,
               &feature_segments, hmat.Ptrs().size(),
               &gidx_fvalue_map, hmat.Values().size(),
               &min_fvalue, hmat.MinValues().size());
  dh::CopyVectorToDeviceSpan(gidx_fvalue_map, hmat.Values());
  dh::CopyVectorToDeviceSpan(min_fvalue, hmat.MinValues());
  dh::CopyVectorToDeviceSpan(feature_segments, hmat.Ptrs());
}

// Initialize the EllpackInfo for this page.
void EllpackPageImpl::InitInfo(int device,
                               bool is_dense,
                               size_t row_stride,
                               const common::HistogramCuts& hmat) {
  matrix.info = EllpackInfo(device, is_dense, row_stride, hmat, &ba_);
}

// Initialize the buffer to stored compressed features.
void EllpackPageImpl::InitCompressedData(int device, size_t num_rows) {
  size_t num_symbols = matrix.info.NumSymbols();

  // Required buffer size for storing data matrix in ELLPack format.
  size_t compressed_size_bytes = common::CompressedBufferWriter::CalculateBufferSize(
      matrix.info.row_stride * num_rows, num_symbols);
  ba_.Allocate(device, &gidx_buffer, compressed_size_bytes);

  thrust::fill(dh::tbegin(gidx_buffer), dh::tend(gidx_buffer), 0);

  matrix.gidx_iter = common::CompressedIterator<uint32_t>(gidx_buffer.data(), num_symbols);
}

// Compress a CSR page into ELLPACK.
void EllpackPageImpl::CreateHistIndices(int device,
                                        const SparsePage& row_batch,
                                        const RowStateOnDevice& device_row_state) {
  // Has any been allocated for me in this batch?
  if (!device_row_state.rows_to_process_from_batch) return;

  unsigned int null_gidx_value = matrix.info.n_bins;
  size_t row_stride = matrix.info.row_stride;

  const auto& offset_vec = row_batch.offset.ConstHostVector();

  // bin and compress entries in batches of rows
  size_t gpu_batch_nrows = std::min(
      dh::TotalMemory(device) / (16 * row_stride * sizeof(Entry)),
      static_cast<size_t>(device_row_state.rows_to_process_from_batch));
  const std::vector<Entry>& data_vec = row_batch.data.ConstHostVector();

  size_t gpu_nbatches = common::DivRoundUp(device_row_state.rows_to_process_from_batch,
                                           gpu_batch_nrows);

  for (size_t gpu_batch = 0; gpu_batch < gpu_nbatches; ++gpu_batch) {
    size_t batch_row_begin = gpu_batch * gpu_batch_nrows;
    size_t batch_row_end = (gpu_batch + 1) * gpu_batch_nrows;
    if (batch_row_end > device_row_state.rows_to_process_from_batch) {
      batch_row_end = device_row_state.rows_to_process_from_batch;
    }
    size_t batch_nrows = batch_row_end - batch_row_begin;

    const auto ent_cnt_begin =
        offset_vec[device_row_state.row_offset_in_current_batch + batch_row_begin];
    const auto ent_cnt_end =
        offset_vec[device_row_state.row_offset_in_current_batch + batch_row_end];

    /*! \brief row offset in SparsePage (the input data). */
    dh::device_vector<size_t> row_ptrs(batch_nrows + 1);
    thrust::copy(
        offset_vec.data() + device_row_state.row_offset_in_current_batch + batch_row_begin,
        offset_vec.data() + device_row_state.row_offset_in_current_batch + batch_row_end + 1,
        row_ptrs.begin());

    // number of entries in this batch.
    size_t n_entries = ent_cnt_end - ent_cnt_begin;
    dh::device_vector<Entry> entries_d(n_entries);
    // copy data entries to device.
    dh::safe_cuda(hipMemcpy(entries_d.data().get(),
                             data_vec.data() + ent_cnt_begin,
                             n_entries * sizeof(Entry),
                             hipMemcpyDefault));
    const dim3 block3(32, 8, 1);  // 256 threads
    const dim3 grid3(common::DivRoundUp(batch_nrows, block3.x),
                     common::DivRoundUp(row_stride, block3.y),
                     1);
    dh::LaunchKernel {grid3, block3} (
        CompressBinEllpackKernel,
        common::CompressedBufferWriter(matrix.info.NumSymbols()),
        gidx_buffer.data(),
        row_ptrs.data().get(),
        entries_d.data().get(),
        matrix.info.gidx_fvalue_map.data(),
        matrix.info.feature_segments.data(),
        device_row_state.total_rows_processed + batch_row_begin,
        batch_nrows,
        row_stride,
        null_gidx_value);
  }
}

// Return the number of rows contained in this page.
size_t EllpackPageImpl::Size() const {
  return matrix.n_rows;
}

// Clear the current page.
void EllpackPageImpl::Clear() {
  ba_.Clear();
  gidx_buffer = {};
  idx_buffer.clear();
  sparse_page_.Clear();
  matrix.base_rowid = 0;
  matrix.n_rows = 0;
  device_initialized_ = false;
}

// Push a CSR page to the current page.
//
// The CSR pages are accumulated in memory until they reach a certain size, then written out as
// compressed ELLPACK.
void EllpackPageImpl::Push(int device, const SparsePage& batch) {
  sparse_page_.Push(batch);
  matrix.n_rows += batch.Size();
}

// Compress the accumulated SparsePage.
void EllpackPageImpl::CompressSparsePage(int device) {
  monitor_.StartCuda("InitCompressedData");
  InitCompressedData(device, matrix.n_rows);
  monitor_.StopCuda("InitCompressedData");

  monitor_.StartCuda("BinningCompression");
  DeviceHistogramBuilderState hist_builder_row_state(matrix.n_rows);
  hist_builder_row_state.BeginBatch(sparse_page_);
  CreateHistIndices(device, sparse_page_, hist_builder_row_state.GetRowStateOnDevice());
  hist_builder_row_state.EndBatch();
  monitor_.StopCuda("BinningCompression");

  monitor_.StartCuda("CopyDeviceToHost");
  idx_buffer.resize(gidx_buffer.size());
  dh::CopyDeviceSpanToVector(&idx_buffer, gidx_buffer);
  ba_.Clear();
  gidx_buffer = {};
  monitor_.StopCuda("CopyDeviceToHost");
}

// Return the memory cost for storing the compressed features.
size_t EllpackPageImpl::MemCostBytes() const {
  // Required buffer size for storing data matrix in ELLPack format.
  size_t compressed_size_bytes = common::CompressedBufferWriter::CalculateBufferSize(
      matrix.info.row_stride * matrix.n_rows, matrix.info.NumSymbols());
  return compressed_size_bytes;
}

// Copy the compressed features to GPU.
void EllpackPageImpl::InitDevice(int device, EllpackInfo info) {
  if (device_initialized_) return;

  monitor_.StartCuda("CopyPageToDevice");
  dh::safe_cuda(hipSetDevice(device));

  gidx_buffer = {};
  ba_.Allocate(device, &gidx_buffer, idx_buffer.size());
  dh::CopyVectorToDeviceSpan(gidx_buffer, idx_buffer);

  matrix.info = info;
  matrix.gidx_iter = common::CompressedIterator<uint32_t>(gidx_buffer.data(), info.n_bins + 1);

  monitor_.StopCuda("CopyPageToDevice");

  device_initialized_ = true;
}
}  // namespace xgboost
