#include "hip/hip_runtime.h"
/*!
 * Copyright 2019 XGBoost contributors
 */

#include <xgboost/data.h>

#include "./ellpack_page.cuh"
#include "../common/hist_util.h"
#include "../common/random.h"

namespace xgboost {

EllpackPage::EllpackPage() : impl_{new EllpackPageImpl()} {}

EllpackPage::EllpackPage(DMatrix* dmat, const BatchParam& param)
    : impl_{new EllpackPageImpl(dmat, param)} {}

EllpackPage::~EllpackPage() = default;

size_t EllpackPage::Size() const {
  return impl_->Size();
}

void EllpackPage::SetBaseRowId(size_t row_id) {
  impl_->SetBaseRowId(row_id);
}

// Bin each input data entry, store the bin indices in compressed form.
__global__ void CompressBinEllpackKernel(
    common::CompressedBufferWriter wr,
    common::CompressedByteT* __restrict__ buffer,  // gidx_buffer
    const size_t* __restrict__ row_ptrs,           // row offset of input data
    const Entry* __restrict__ entries,      // One batch of input data
    const float* __restrict__ cuts,         // HistogramCuts::cut
    const uint32_t* __restrict__ cut_rows,  // HistogramCuts::row_ptrs
    size_t base_row,                        // batch_row_begin
    size_t n_rows,
    size_t row_stride,
    unsigned int null_gidx_value) {
  size_t irow = threadIdx.x + blockIdx.x * blockDim.x;
  int ifeature = threadIdx.y + blockIdx.y * blockDim.y;
  if (irow >= n_rows || ifeature >= row_stride) {
    return;
  }
  int row_length = static_cast<int>(row_ptrs[irow + 1] - row_ptrs[irow]);
  unsigned int bin = null_gidx_value;
  if (ifeature < row_length) {
    Entry entry = entries[row_ptrs[irow] - row_ptrs[0] + ifeature];
    int feature = entry.index;
    float fvalue = entry.fvalue;
    // {feature_cuts, ncuts} forms the array of cuts of `feature'.
    const float* feature_cuts = &cuts[cut_rows[feature]];
    int ncuts = cut_rows[feature + 1] - cut_rows[feature];
    // Assigning the bin in current entry.
    // S.t.: fvalue < feature_cuts[bin]
    bin = dh::UpperBound(feature_cuts, ncuts, fvalue);
    if (bin >= ncuts) {
      bin = ncuts - 1;
    }
    // Add the number of bins in previous features.
    bin += cut_rows[feature];
  }
  // Write to gidx buffer.
  wr.AtomicWriteSymbol(buffer, bin, (irow + base_row) * row_stride + ifeature);
}

// Construct an ELLPACK matrix in memory.
EllpackPageImpl::EllpackPageImpl(DMatrix* dmat, const BatchParam& param) {
  monitor_.Init("ellpack_page");
  dh::safe_cuda(hipSetDevice(param.gpu_id));

  matrix.n_rows = dmat->Info().num_row_;

  monitor_.StartCuda("Quantiles");
  // Create the quantile sketches for the dmatrix and initialize HistogramCuts.
  common::HistogramCuts hmat;
  size_t row_stride =
      common::DeviceSketch(param.gpu_id, param.max_bin, param.gpu_batch_nrows, dmat, &hmat);
  monitor_.StopCuda("Quantiles");

  monitor_.StartCuda("InitEllpackInfo");
  InitInfo(param.gpu_id, dmat->IsDense(), row_stride, hmat);
  monitor_.StopCuda("InitEllpackInfo");

  monitor_.StartCuda("InitCompressedData");
  InitCompressedData(param.gpu_id, dmat->Info().num_row_);
  monitor_.StopCuda("InitCompressedData");

  monitor_.StartCuda("BinningCompression");
  DeviceHistogramBuilderState hist_builder_row_state(dmat->Info().num_row_);
  for (const auto& batch : dmat->GetBatches<SparsePage>()) {
    hist_builder_row_state.BeginBatch(batch);
    CreateHistIndices(param.gpu_id, batch, hist_builder_row_state.GetRowStateOnDevice());
    hist_builder_row_state.EndBatch();
  }
  monitor_.StopCuda("BinningCompression");
}

// Construct an EllpackInfo based on histogram cuts of features.
EllpackInfo::EllpackInfo(int device,
                         bool is_dense,
                         size_t row_stride,
                         const common::HistogramCuts& hmat,
                         dh::BulkAllocator* ba)
    : is_dense(is_dense), row_stride(row_stride), n_bins(hmat.Ptrs().back()) {

  ba->Allocate(device,
               &feature_segments, hmat.Ptrs().size(),
               &gidx_fvalue_map, hmat.Values().size(),
               &min_fvalue, hmat.MinValues().size());
  dh::CopyVectorToDeviceSpan(gidx_fvalue_map, hmat.Values());
  dh::CopyVectorToDeviceSpan(min_fvalue, hmat.MinValues());
  dh::CopyVectorToDeviceSpan(feature_segments, hmat.Ptrs());
}

// Initialize the EllpackInfo for this page.
void EllpackPageImpl::InitInfo(int device,
                               bool is_dense,
                               size_t row_stride,
                               const common::HistogramCuts& hmat) {
  matrix.info = EllpackInfo(device, is_dense, row_stride, hmat, &ba_);
}

// Initialize the buffer to stored compressed features.
void EllpackPageImpl::InitCompressedData(int device, size_t num_rows) {
  size_t num_symbols = matrix.info.n_bins + 1;

  // Required buffer size for storing data matrix in ELLPack format.
  size_t compressed_size_bytes = common::CompressedBufferWriter::CalculateBufferSize(
      matrix.info.row_stride * num_rows, num_symbols);
  ba_.Allocate(device, &gidx_buffer, compressed_size_bytes);

  thrust::fill(
      thrust::device_pointer_cast(gidx_buffer.data()),
      thrust::device_pointer_cast(gidx_buffer.data() + gidx_buffer.size()), 0);

  matrix.gidx_iter = common::CompressedIterator<uint32_t>(gidx_buffer.data(), num_symbols);
}

// Compress a CSR page into ELLPACK.
void EllpackPageImpl::CreateHistIndices(int device,
                                        const SparsePage& row_batch,
                                        const RowStateOnDevice& device_row_state) {
  // Has any been allocated for me in this batch?
  if (!device_row_state.rows_to_process_from_batch) return;

  unsigned int null_gidx_value = matrix.info.n_bins;
  size_t row_stride = matrix.info.row_stride;

  const auto& offset_vec = row_batch.offset.ConstHostVector();

  int num_symbols = matrix.info.n_bins + 1;
  // bin and compress entries in batches of rows
  size_t gpu_batch_nrows = std::min(
      dh::TotalMemory(device) / (16 * row_stride * sizeof(Entry)),
      static_cast<size_t>(device_row_state.rows_to_process_from_batch));
  const std::vector<Entry>& data_vec = row_batch.data.ConstHostVector();

  size_t gpu_nbatches = common::DivRoundUp(device_row_state.rows_to_process_from_batch,
                                           gpu_batch_nrows);

  for (size_t gpu_batch = 0; gpu_batch < gpu_nbatches; ++gpu_batch) {
    size_t batch_row_begin = gpu_batch * gpu_batch_nrows;
    size_t batch_row_end = (gpu_batch + 1) * gpu_batch_nrows;
    if (batch_row_end > device_row_state.rows_to_process_from_batch) {
      batch_row_end = device_row_state.rows_to_process_from_batch;
    }
    size_t batch_nrows = batch_row_end - batch_row_begin;

    const auto ent_cnt_begin =
        offset_vec[device_row_state.row_offset_in_current_batch + batch_row_begin];
    const auto ent_cnt_end =
        offset_vec[device_row_state.row_offset_in_current_batch + batch_row_end];

    /*! \brief row offset in SparsePage (the input data). */
    dh::device_vector<size_t> row_ptrs(batch_nrows + 1);
    thrust::copy(
        offset_vec.data() + device_row_state.row_offset_in_current_batch + batch_row_begin,
        offset_vec.data() + device_row_state.row_offset_in_current_batch + batch_row_end + 1,
        row_ptrs.begin());

    // number of entries in this batch.
    size_t n_entries = ent_cnt_end - ent_cnt_begin;
    dh::device_vector<Entry> entries_d(n_entries);
    // copy data entries to device.
    dh::safe_cuda(hipMemcpy(entries_d.data().get(),
                             data_vec.data() + ent_cnt_begin,
                             n_entries * sizeof(Entry),
                             hipMemcpyDefault));
    const dim3 block3(32, 8, 1);  // 256 threads
    const dim3 grid3(common::DivRoundUp(batch_nrows, block3.x),
                     common::DivRoundUp(row_stride, block3.y),
                     1);
    dh::LaunchKernel {grid3, block3} (
        CompressBinEllpackKernel,
        common::CompressedBufferWriter(num_symbols),
        gidx_buffer.data(),
        row_ptrs.data().get(),
        entries_d.data().get(),
        matrix.info.gidx_fvalue_map.data(),
        matrix.info.feature_segments.data(),
        device_row_state.total_rows_processed + batch_row_begin,
        batch_nrows,
        row_stride,
        null_gidx_value);
  }
}

// Return the number of rows contained in this page.
size_t EllpackPageImpl::Size() const {
  return matrix.n_rows;
}

// Clear the current page.
void EllpackPageImpl::Clear() {
  ba_.Clear();
  gidx_buffer = {};
  idx_buffer.clear();
  sparse_page_.Clear();
  matrix.base_rowid = 0;
  matrix.n_rows = 0;
  device_initialized_ = false;
}

// Push a CSR page to the current page.
//
// The CSR pages are accumulated in memory until they reach a certain size, then written out as
// compressed ELLPACK.
void EllpackPageImpl::Push(int device, const SparsePage& batch) {
  sparse_page_.Push(batch);
  matrix.n_rows += batch.Size();
}

// Compress the accumulated SparsePage.
void EllpackPageImpl::CompressSparsePage(int device) {
  monitor_.StartCuda("InitCompressedData");
  InitCompressedData(device, matrix.n_rows);
  monitor_.StopCuda("InitCompressedData");

  monitor_.StartCuda("BinningCompression");
  DeviceHistogramBuilderState hist_builder_row_state(matrix.n_rows);
  hist_builder_row_state.BeginBatch(sparse_page_);
  CreateHistIndices(device, sparse_page_, hist_builder_row_state.GetRowStateOnDevice());
  hist_builder_row_state.EndBatch();
  monitor_.StopCuda("BinningCompression");

  monitor_.StartCuda("CopyDeviceToHost");
  idx_buffer.resize(gidx_buffer.size());
  dh::CopyDeviceSpanToVector(&idx_buffer, gidx_buffer);
  ba_.Clear();
  gidx_buffer = {};
  monitor_.StopCuda("CopyDeviceToHost");
}

// Return the memory cost for storing the compressed features.
size_t EllpackPageImpl::MemCostBytes() const {
  size_t num_symbols = matrix.info.n_bins + 1;

  // Required buffer size for storing data matrix in ELLPack format.
  size_t compressed_size_bytes = common::CompressedBufferWriter::CalculateBufferSize(
      matrix.info.row_stride * matrix.n_rows, num_symbols);
  return compressed_size_bytes;
}

// Copy the compressed features to GPU.
void EllpackPageImpl::InitDevice(int device, EllpackInfo info) {
  if (device_initialized_) return;

  monitor_.StartCuda("CopyPageToDevice");
  dh::safe_cuda(hipSetDevice(device));

  gidx_buffer = {};
  ba_.Allocate(device, &gidx_buffer, idx_buffer.size());
  dh::CopyVectorToDeviceSpan(gidx_buffer, idx_buffer);

  matrix.info = info;
  matrix.gidx_iter = common::CompressedIterator<uint32_t>(gidx_buffer.data(), info.n_bins + 1);

  monitor_.StopCuda("CopyPageToDevice");

  device_initialized_ = true;
}

}  // namespace xgboost
