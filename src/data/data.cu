/*!
 * Copyright 2019-2021 by XGBoost Contributors
 *
 * \file data.cu
 * \brief Handles setting metainfo from array interface.
 */
#include "xgboost/data.h"
#include "xgboost/logging.h"
#include "xgboost/json.h"
#include "array_interface.h"
#include "../common/device_helpers.cuh"
#include "device_adapter.cuh"
#include "simple_dmatrix.h"

namespace xgboost {

void CopyInfoImpl(ArrayInterface column, HostDeviceVector<float>* out) {
  auto SetDeviceToPtr = [](void* ptr) {
    hipPointerAttribute_t attr;
    dh::safe_cuda(hipPointerGetAttributes(&attr, ptr));
    int32_t ptr_device = attr.device;
    dh::safe_cuda(hipSetDevice(ptr_device));
    return ptr_device;
  };
  auto ptr_device = SetDeviceToPtr(column.data);

  out->SetDevice(ptr_device);
  out->Resize(column.num_rows);

  auto p_dst = thrust::device_pointer_cast(out->DevicePointer());

  dh::LaunchN(column.num_rows, [=] __device__(size_t idx) {
    p_dst[idx] = column.GetElement(idx, 0);
  });
}

namespace {
auto SetDeviceToPtr(void *ptr) {
  hipPointerAttribute_t attr;
  dh::safe_cuda(hipPointerGetAttributes(&attr, ptr));
  int32_t ptr_device = attr.device;
  dh::safe_cuda(hipSetDevice(ptr_device));
  return ptr_device;
}
}  // anonymous namespace

void CopyGroupInfoImpl(ArrayInterface column, std::vector<bst_group_t>* out) {
  CHECK(column.type != ArrayInterface::kF4 && column.type != ArrayInterface::kF8)
      << "Expected integer for group info.";

  auto ptr_device = SetDeviceToPtr(column.data);
  CHECK_EQ(ptr_device, dh::CurrentDevice());
  dh::TemporaryArray<bst_group_t> temp(column.num_rows);
  auto d_tmp = temp.data();

  dh::LaunchN(column.num_rows, [=] __device__(size_t idx) {
    d_tmp[idx] = column.GetElement<size_t>(idx, 0);
  });
  auto length = column.num_rows;
  out->resize(length + 1);
  out->at(0) = 0;
  thrust::copy(temp.data(), temp.data() + length, out->begin() + 1);
  std::partial_sum(out->begin(), out->end(), out->begin());
}

void CopyQidImpl(ArrayInterface array_interface,
                 std::vector<bst_group_t> *p_group_ptr) {
  auto &group_ptr_ = *p_group_ptr;
  auto it = dh::MakeTransformIterator<uint32_t>(
      thrust::make_counting_iterator(0ul),
      [array_interface] __device__(size_t i) {
        return array_interface.GetElement<uint32_t>(i, 0);
      });
  dh::caching_device_vector<bool> flag(1);
  auto d_flag = dh::ToSpan(flag);
  auto d = SetDeviceToPtr(array_interface.data);
  dh::LaunchN(1, [=] __device__(size_t) { d_flag[0] = true; });
  dh::LaunchN(array_interface.num_rows - 1, [=] __device__(size_t i) {
    if (array_interface.GetElement<uint32_t>(i, 0) >
        array_interface.GetElement<uint32_t>(i + 1, 0)) {
      d_flag[0] = false;
    }
  });
  bool non_dec = true;
  dh::safe_cuda(hipMemcpy(&non_dec, flag.data().get(), sizeof(bool),
                           hipMemcpyDeviceToHost));
  CHECK(non_dec) << "`qid` must be sorted in increasing order along with data.";
  size_t bytes = 0;
  dh::caching_device_vector<uint32_t> out(array_interface.num_rows);
  dh::caching_device_vector<uint32_t> cnt(array_interface.num_rows);
  HostDeviceVector<int> d_num_runs_out(1, 0, d);
  hipcub::DeviceRunLengthEncode::Encode(
      nullptr, bytes, it, out.begin(), cnt.begin(),
      d_num_runs_out.DevicePointer(), array_interface.num_rows);
  dh::caching_device_vector<char> tmp(bytes);
  hipcub::DeviceRunLengthEncode::Encode(
      tmp.data().get(), bytes, it, out.begin(), cnt.begin(),
      d_num_runs_out.DevicePointer(), array_interface.num_rows);

  auto h_num_runs_out = d_num_runs_out.HostSpan()[0];
  group_ptr_.clear();
  group_ptr_.resize(h_num_runs_out + 1, 0);
  dh::XGBCachingDeviceAllocator<char> alloc;
  thrust::inclusive_scan(thrust::cuda::par(alloc), cnt.begin(),
                         cnt.begin() + h_num_runs_out, cnt.begin());
  thrust::copy(cnt.begin(), cnt.begin() + h_num_runs_out,
               group_ptr_.begin() + 1);
}

namespace {
// thrust::all_of tries to copy lambda function.
struct AllOfOp {
  __device__ bool operator()(float w) {
    return w >= 0;
  }
};
}  // anonymous namespace

void MetaInfo::SetInfo(const char * c_key, std::string const& interface_str) {
  Json j_interface = Json::Load({interface_str.c_str(), interface_str.size()});
  auto const& j_arr = get<Array>(j_interface);
  CHECK_EQ(j_arr.size(), 1)
      << "MetaInfo: " << c_key << ". " << ArrayInterfaceErrors::Dimension(1);
  ArrayInterface array_interface(interface_str);
  std::string key{c_key};
  array_interface.AsColumnVector();
  CHECK(!array_interface.valid.Data())
      << "Meta info " << key << " should be dense, found validity mask";
  if (array_interface.num_rows == 0) {
    return;
  }

  if (key == "label") {
    CopyInfoImpl(array_interface, &labels_);
  } else if (key == "weight") {
    CopyInfoImpl(array_interface, &weights_);
    auto ptr = weights_.ConstDevicePointer();
    auto valid =
        thrust::all_of(thrust::device, ptr, ptr + weights_.Size(), AllOfOp{});
    CHECK(valid) << "Weights must be positive values.";
  } else if (key == "base_margin") {
    CopyInfoImpl(array_interface, &base_margin_);
  } else if (key == "group") {
    CopyGroupInfoImpl(array_interface, &group_ptr_);
    return;
  } else if (key == "qid") {
    CopyQidImpl(array_interface, &group_ptr_);
    return;
  } else if (key == "label_lower_bound") {
    CopyInfoImpl(array_interface, &labels_lower_bound_);
    return;
  } else if (key == "label_upper_bound") {
    CopyInfoImpl(array_interface, &labels_upper_bound_);
    return;
  } else if (key == "feature_weights") {
    CopyInfoImpl(array_interface, &feature_weigths);
    auto d_feature_weights = feature_weigths.ConstDeviceSpan();
    auto valid = thrust::all_of(
        thrust::device, d_feature_weights.data(),
        d_feature_weights.data() + d_feature_weights.size(), AllOfOp{});
    CHECK(valid) << "Feature weight must be greater than 0.";
    return;
  } else {
    LOG(FATAL) << "Unknown metainfo: " << key;
  }
}

template <typename AdapterT>
DMatrix* DMatrix::Create(AdapterT* adapter, float missing, int nthread,
                         const std::string& cache_prefix) {
  CHECK_EQ(cache_prefix.size(), 0)
      << "Device memory construction is not currently supported with external "
         "memory.";
  return new data::SimpleDMatrix(adapter, missing, nthread);
}

template DMatrix* DMatrix::Create<data::CudfAdapter>(
    data::CudfAdapter* adapter, float missing, int nthread,
    const std::string& cache_prefix);
template DMatrix* DMatrix::Create<data::CupyAdapter>(
    data::CupyAdapter* adapter, float missing, int nthread,
    const std::string& cache_prefix);
}  // namespace xgboost
