/*!
 * Copyright 2019 by XGBoost Contributors
 *
 * \file data.cu
 * \brief Handles setting metainfo from array interface.
 */
#include "xgboost/data.h"
#include "xgboost/logging.h"

#include "columnar.h"
#include "../common/device_helpers.cuh"
#include "../common/json_experimental.h"
#include "../common/json_reader_experimental.h"

namespace xgboost {

template <typename T>
void CopyInfoImpl(experimental::Json const& column, HostDeviceVector<float>* out) {
  auto SetDeviceToPtr = [](void* ptr) {
    hipPointerAttribute_t attr;
    dh::safe_cuda(hipPointerGetAttributes(&attr, ptr));
    int32_t ptr_device = attr.device;
    dh::safe_cuda(hipSetDevice(ptr_device));
    return ptr_device;
  };

  common::Span<T> s_data { ArrayInterfaceHandler::ExtractData<T>(column) };
  auto ptr_device = SetDeviceToPtr(s_data.data());
  thrust::device_ptr<T> p_src {s_data.data()};

  auto length = s_data.size();
  out->SetDevice(ptr_device);
  out->Resize(length);

  auto p_dst = thrust::device_pointer_cast(out->DevicePointer());
  thrust::copy(p_src, p_src + length, p_dst);
}

void MetaInfo::SetInfo(const char * c_key, std::string const& interface_str) {
  std::string copied = interface_str;
  experimental::Document j_interface =
      experimental::Document::Load<experimental::JsonRecursiveReader>(
          experimental::StringRef{copied});
  auto const& j_arr = j_interface.GetValue();
  CHECK_EQ(j_arr.Length(), 1) << "MetaInfo: " << c_key << ". " << ColumnarErrors::Dimension(1);;
  auto const& j_arr_obj = j_arr.GetArrayElem(0);
  std::string key {c_key};
  ArrayInterfaceHandler::Validate(j_arr_obj);
  if (j_arr_obj.FindMemberByKey("mask") != j_arr_obj.cend()) {
    LOG(FATAL) << "Meta info " << key << " should be dense, found validity mask";
  }
  auto const& typestr = (*j_arr_obj.FindMemberByKey("typestr")).GetString();

  if (key == "root_index") {
    LOG(FATAL) << "root index for columnar data is not supported.";
  } else if (key == "label") {
    DISPATCH_TYPE(CopyInfoImpl, typestr, j_arr_obj, &labels_);
  } else if (key == "weight") {
    DISPATCH_TYPE(CopyInfoImpl, typestr, j_arr_obj, &weights_);
  } else if (key == "base_margin") {
    DISPATCH_TYPE(CopyInfoImpl, typestr, j_arr_obj, &base_margin_);
  } else if (key == "group") {
    // Ranking is not performed on device.
    auto s_data = ArrayInterfaceHandler::ExtractData<uint32_t>(j_arr_obj);
    thrust::device_ptr<uint32_t> p_src {s_data.data()};

    auto length = s_data.size();
    group_ptr_.resize(length + 1);
    group_ptr_[0] = 0;
    thrust::copy(p_src, p_src + length, group_ptr_.begin() + 1);
    std::partial_sum(group_ptr_.begin(), group_ptr_.end(), group_ptr_.begin());

    return;
  } else {
    LOG(FATAL) << "Unknown metainfo: " << key;
  }
}
}  // namespace xgboost
