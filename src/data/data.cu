/*!
 * Copyright 2019 by XGBoost Contributors
 *
 * \file data.cu
 * \brief Handles setting metainfo from array interface.
 */
#include "xgboost/data.h"
#include "xgboost/logging.h"
#include "xgboost/json.h"
#include "array_interface.h"
#include "../common/device_helpers.cuh"
#include "device_adapter.cuh"
#include "simple_dmatrix.h"

namespace xgboost {

void CopyInfoImpl(ArrayInterface column, HostDeviceVector<float>* out) {
  auto SetDeviceToPtr = [](void* ptr) {
    hipPointerAttribute_t attr;
    dh::safe_cuda(hipPointerGetAttributes(&attr, ptr));
    int32_t ptr_device = attr.device;
    dh::safe_cuda(hipSetDevice(ptr_device));
    return ptr_device;
  };
  auto ptr_device = SetDeviceToPtr(column.data);

  out->SetDevice(ptr_device);
  out->Resize(column.num_rows);

  auto p_dst = thrust::device_pointer_cast(out->DevicePointer());

  dh::LaunchN(ptr_device, column.num_rows, [=] __device__(size_t idx) {
    p_dst[idx] = column.GetElement(idx);
  });
}

void MetaInfo::SetInfo(const char * c_key, std::string const& interface_str) {
  Json j_interface = Json::Load({interface_str.c_str(), interface_str.size()});
  auto const& j_arr = get<Array>(j_interface);
  CHECK_EQ(j_arr.size(), 1)
      << "MetaInfo: " << c_key << ". " << ArrayInterfaceErrors::Dimension(1);
  ArrayInterface array_interface(get<Object const>(j_arr[0]));
  std::string key{c_key};
  CHECK(!array_interface.valid.Data())
      << "Meta info " << key << " should be dense, found validity mask";
  CHECK_EQ(array_interface.num_cols, 1)
      << "Meta info should be a single column.";

  if (key == "label") {
    CopyInfoImpl(array_interface, &labels_);
  } else if (key == "weight") {
    CopyInfoImpl(array_interface, &weights_);
  } else if (key == "base_margin") {
    CopyInfoImpl(array_interface, &base_margin_);
  } else if (key == "group") {
    // Ranking is not performed on device.
    thrust::device_ptr<uint32_t> p_src{
        reinterpret_cast<uint32_t*>(array_interface.data)};

    auto length = array_interface.num_rows;
    group_ptr_.resize(length + 1);
    group_ptr_[0] = 0;
    thrust::copy(p_src, p_src + length, group_ptr_.begin() + 1);
    std::partial_sum(group_ptr_.begin(), group_ptr_.end(), group_ptr_.begin());

    return;
  } else {
    LOG(FATAL) << "Unknown metainfo: " << key;
  }
}

template <typename AdapterT>
DMatrix* DMatrix::Create(AdapterT* adapter, float missing, int nthread,
                         const std::string& cache_prefix, size_t page_size) {
  CHECK_EQ(cache_prefix.size(), 0)
      << "Device memory construction is not currently supported with external "
         "memory.";
  return new data::SimpleDMatrix(adapter, missing, nthread);
}

template DMatrix* DMatrix::Create<data::CudfAdapter>(
    data::CudfAdapter* adapter, float missing, int nthread,
    const std::string& cache_prefix, size_t page_size);
template DMatrix* DMatrix::Create<data::CupyAdapter>(
    data::CupyAdapter* adapter, float missing, int nthread,
    const std::string& cache_prefix, size_t page_size);
}  // namespace xgboost
