/*!
 * Copyright 2019 by XGBoost Contributors
 *
 * \file data.cu
 * \brief Handles setting metainfo from array interface.
 */
#include "xgboost/data.h"
#include "xgboost/logging.h"
#include "xgboost/json.h"
#include "array_interface.h"
#include "../common/device_helpers.cuh"
#include "device_adapter.cuh"
#include "simple_dmatrix.h"

namespace xgboost {

void CopyInfoImpl(ArrayInterface column, HostDeviceVector<float>* out) {
  auto SetDeviceToPtr = [](void* ptr) {
    hipPointerAttribute_t attr;
    dh::safe_cuda(hipPointerGetAttributes(&attr, ptr));
    int32_t ptr_device = attr.device;
    dh::safe_cuda(hipSetDevice(ptr_device));
    return ptr_device;
  };
  auto ptr_device = SetDeviceToPtr(column.data);

  out->SetDevice(ptr_device);
  out->Resize(column.num_rows);

  auto p_dst = thrust::device_pointer_cast(out->DevicePointer());

  dh::LaunchN(ptr_device, column.num_rows, [=] __device__(size_t idx) {
    p_dst[idx] = column.GetElement(idx);
  });
}

void CopyGroupInfoImpl(ArrayInterface column, std::vector<bst_group_t>* out) {
  CHECK(column.type[1] == 'i' || column.type[1] == 'u')
      << "Expected integer metainfo";
  auto SetDeviceToPtr = [](void* ptr) {
    hipPointerAttribute_t attr;
    dh::safe_cuda(hipPointerGetAttributes(&attr, ptr));
    int32_t ptr_device = attr.device;
    dh::safe_cuda(hipSetDevice(ptr_device));
    return ptr_device;
  };
  auto ptr_device = SetDeviceToPtr(column.data);
  dh::TemporaryArray<bst_group_t> temp(column.num_rows);
  auto d_tmp = temp.data();

  dh::LaunchN(ptr_device, column.num_rows, [=] __device__(size_t idx) {
    d_tmp[idx] = column.GetElement(idx);
  });
  auto length = column.num_rows;
  out->resize(length + 1);
  out->at(0) = 0;
  thrust::copy(temp.data(), temp.data() + length, out->begin() + 1);
  std::partial_sum(out->begin(), out->end(), out->begin());
}

void MetaInfo::SetInfo(const char * c_key, std::string const& interface_str) {
  Json j_interface = Json::Load({interface_str.c_str(), interface_str.size()});
  auto const& j_arr = get<Array>(j_interface);
  CHECK_EQ(j_arr.size(), 1)
      << "MetaInfo: " << c_key << ". " << ArrayInterfaceErrors::Dimension(1);
  ArrayInterface array_interface(interface_str);
  std::string key{c_key};
  CHECK(!array_interface.valid.Data())
      << "Meta info " << key << " should be dense, found validity mask";
  CHECK_EQ(array_interface.num_cols, 1)
      << "Meta info should be a single column.";
  if (array_interface.num_rows == 0) {
    return;
  }

  if (key == "label") {
    CopyInfoImpl(array_interface, &labels_);
  } else if (key == "weight") {
    CopyInfoImpl(array_interface, &weights_);
  } else if (key == "base_margin") {
    CopyInfoImpl(array_interface, &base_margin_);
  } else if (key == "group") {
    CopyGroupInfoImpl(array_interface, &group_ptr_);
    return;
  } else {
    LOG(FATAL) << "Unknown metainfo: " << key;
  }
}

template <typename AdapterT>
DMatrix* DMatrix::Create(AdapterT* adapter, float missing, int nthread,
                         const std::string& cache_prefix, size_t page_size) {
  CHECK_EQ(cache_prefix.size(), 0)
      << "Device memory construction is not currently supported with external "
         "memory.";
  return new data::SimpleDMatrix(adapter, missing, nthread);
}

template DMatrix* DMatrix::Create<data::CudfAdapter>(
    data::CudfAdapter* adapter, float missing, int nthread,
    const std::string& cache_prefix, size_t page_size);
template DMatrix* DMatrix::Create<data::CupyAdapter>(
    data::CupyAdapter* adapter, float missing, int nthread,
    const std::string& cache_prefix, size_t page_size);
}  // namespace xgboost
