/**
 * Copyright 2019-2024, XGBoost contributors
 */
#include <dmlc/registry.h>

#include <cstddef>  // for size_t
#include <cstdint>  // for uint64_t

#include "../common/io.h"                 // for AlignedResourceReadStream, AlignedFileWriteStream
#include "../common/ref_resource_view.h"  // for ReadVec, WriteVec
#include "ellpack_page.cuh"               // for EllpackPage
#include "ellpack_page_raw_format.h"
#include "ellpack_page_source.h"

namespace xgboost::data {
DMLC_REGISTRY_FILE_TAG(ellpack_page_raw_format);

namespace {
template <typename T>
[[nodiscard]] bool ReadDeviceVec(common::AlignedResourceReadStream* fi, HostDeviceVector<T>* vec) {
  std::uint64_t n{0};
  if (!fi->Read(&n)) {
    return false;
  }
  if (n == 0) {
    return true;
  }

  auto expected_bytes = sizeof(T) * n;

  auto [ptr, n_bytes] = fi->Consume(expected_bytes);
  if (n_bytes != expected_bytes) {
    return false;
  }

  vec->Resize(n);
  auto d_vec = vec->DeviceSpan();
  dh::safe_cuda(
      hipMemcpyAsync(d_vec.data(), ptr, n_bytes, hipMemcpyDefault, dh::DefaultStream()));
  return true;
}
}  // namespace

[[nodiscard]] bool EllpackPageRawFormat::Read(EllpackPage* page,
                                              common::AlignedResourceReadStream* fi) {
  auto* impl = page->Impl();
  impl->SetCuts(this->cuts_);
  if (!fi->Read(&impl->n_rows)) {
    return false;
  }
  if (!fi->Read(&impl->is_dense)) {
    return false;
  }
  if (!fi->Read(&impl->row_stride)) {
    return false;
  }
  impl->gidx_buffer.SetDevice(device_);
  if (!ReadDeviceVec(fi, &impl->gidx_buffer)) {
    return false;
  }
  if (!fi->Read(&impl->base_rowid)) {
    return false;
  }
  return true;
}

[[nodiscard]] std::size_t EllpackPageRawFormat::Write(const EllpackPage& page,
                                                      common::AlignedFileWriteStream* fo) {
  std::size_t bytes{0};
  auto* impl = page.Impl();
  bytes += fo->Write(impl->n_rows);
  bytes += fo->Write(impl->is_dense);
  bytes += fo->Write(impl->row_stride);
  CHECK(!impl->gidx_buffer.ConstHostVector().empty());
  bytes += common::WriteVec(fo, impl->gidx_buffer.HostVector());
  bytes += fo->Write(impl->base_rowid);
  dh::DefaultStream().Sync();
  return bytes;
}

[[nodiscard]] bool EllpackPageRawFormat::Read(EllpackPage* page, EllpackHostCacheStream* fi) const {
  auto* impl = page->Impl();
  CHECK(this->cuts_->cut_values_.DeviceCanRead());
  impl->SetCuts(this->cuts_);
  if (!fi->Read(&impl->n_rows)) {
    return false;
  }
  if (!fi->Read(&impl->is_dense)) {
    return false;
  }
  if (!fi->Read(&impl->row_stride)) {
    return false;
  }

  // Read vec
  bst_idx_t n{0};
  if (!fi->Read(&n)) {
    return false;
  }
  if (n != 0) {
    impl->gidx_buffer.SetDevice(device_);
    impl->gidx_buffer.Resize(n);
    auto span = impl->gidx_buffer.DeviceSpan();
    if (!fi->Read(span.data(), span.size_bytes())) {
      return false;
    }
  }

  if (!fi->Read(&impl->base_rowid)) {
    return false;
  }

  dh::DefaultStream().Sync();
  return true;
}

[[nodiscard]] std::size_t EllpackPageRawFormat::Write(const EllpackPage& page,
                                                      EllpackHostCacheStream* fo) const {
  bst_idx_t bytes{0};
  auto* impl = page.Impl();
  bytes += fo->Write(impl->n_rows);
  bytes += fo->Write(impl->is_dense);
  bytes += fo->Write(impl->row_stride);

  // Write vector
  bst_idx_t n = impl->gidx_buffer.Size();
  bytes += fo->Write(n);

  if (!impl->gidx_buffer.Empty()) {
    auto span = impl->gidx_buffer.ConstDeviceSpan();
    bytes += fo->Write(span.data(), span.size_bytes());
  }
  bytes += fo->Write(impl->base_rowid);

  dh::DefaultStream().Sync();
  return bytes;
}
}  // namespace xgboost::data
