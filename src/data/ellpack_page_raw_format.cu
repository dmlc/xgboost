/**
 * Copyright 2019-2024, XGBoost contributors
 */
#include <dmlc/registry.h>

#include <cstddef>  // for size_t
#include <cstdint>  // for uint64_t

#include "../common/io.h"                 // for AlignedResourceReadStream, AlignedFileWriteStream
#include "../common/ref_resource_view.h"  // for ReadVec, WriteVec
#include "ellpack_page.cuh"               // for EllpackPage
#include "ellpack_page_raw_format.h"

namespace xgboost::data {
DMLC_REGISTRY_FILE_TAG(ellpack_page_raw_format);

namespace {
template <typename T>
[[nodiscard]] bool ReadDeviceVec(common::AlignedResourceReadStream* fi, HostDeviceVector<T>* vec) {
  std::uint64_t n{0};
  if (!fi->Read(&n)) {
    return false;
  }
  if (n == 0) {
    return true;
  }

  auto expected_bytes = sizeof(T) * n;

  auto [ptr, n_bytes] = fi->Consume(expected_bytes);
  if (n_bytes != expected_bytes) {
    return false;
  }

  vec->SetDevice(DeviceOrd::CUDA(0));
  vec->Resize(n);
  auto d_vec = vec->DeviceSpan();
  dh::safe_cuda(
      hipMemcpyAsync(d_vec.data(), ptr, n_bytes, hipMemcpyDefault, dh::DefaultStream()));
  return true;
}
}  // namespace

[[nodiscard]] bool EllpackPageRawFormat::Read(EllpackPage* page,
                                              common::AlignedResourceReadStream* fi) {
  auto* impl = page->Impl();
  impl->SetCuts(this->cuts_);
  if (!fi->Read(&impl->n_rows)) {
    return false;
  }
  if (!fi->Read(&impl->is_dense)) {
    return false;
  }
  if (!fi->Read(&impl->row_stride)) {
    return false;
  }
  if (!ReadDeviceVec(fi, &impl->gidx_buffer)) {
    return false;
  }
  if (!fi->Read(&impl->base_rowid)) {
    return false;
  }
  return true;
}

[[nodiscard]] std::size_t EllpackPageRawFormat::Write(const EllpackPage& page,
                                                      common::AlignedFileWriteStream* fo) {
  std::size_t bytes{0};
  auto* impl = page.Impl();
  bytes += fo->Write(impl->n_rows);
  bytes += fo->Write(impl->is_dense);
  bytes += fo->Write(impl->row_stride);
  CHECK(!impl->gidx_buffer.ConstHostVector().empty());
  bytes += common::WriteVec(fo, impl->gidx_buffer.HostVector());
  bytes += fo->Write(impl->base_rowid);
  return bytes;
}
}  // namespace xgboost::data
