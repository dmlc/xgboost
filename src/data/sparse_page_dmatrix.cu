/**
 * Copyright 2021-2023 by XGBoost contributors
 */
#include <memory>  // for unique_ptr

#include "../common/hist_util.cuh"
#include "../common/hist_util.h"  // for HistogramCuts
#include "batch_utils.h"          // for CheckEmpty, RegenGHist
#include "ellpack_page.cuh"
#include "sparse_page_dmatrix.h"
#include "xgboost/context.h"  // for Context
#include "xgboost/data.h"     // for BatchParam

namespace xgboost::data {
BatchSet<EllpackPage> SparsePageDMatrix::GetEllpackBatches(Context const* ctx,
                                                           const BatchParam& param) {
  CHECK(ctx->IsCUDA());
  if (param.Initialized()) {
    CHECK_GE(param.max_bin, 2);
  }
  detail::CheckEmpty(batch_param_, param);
  auto id = MakeCache(this, ".ellpack.page", cache_prefix_, &cache_info_);
  size_t row_stride = 0;
  this->InitializeSparsePage(ctx);
  if (!cache_info_.at(id)->written || detail::RegenGHist(batch_param_, param)) {
    // reinitialize the cache
    cache_info_.erase(id);
    MakeCache(this, ".ellpack.page", cache_prefix_, &cache_info_);
    std::unique_ptr<common::HistogramCuts> cuts;
    if (!param.hess.empty()) {
      cuts = std::make_unique<common::HistogramCuts>(
          common::DeviceSketchWithHessian(ctx, this, param.max_bin, param.hess));
    } else {
      cuts =
          std::make_unique<common::HistogramCuts>(common::DeviceSketch(ctx, this, param.max_bin));
    }
    this->InitializeSparsePage(ctx);  // reset after use.

    row_stride = GetRowStride(this);
    this->InitializeSparsePage(ctx);  // reset after use.
    CHECK_NE(row_stride, 0);
    batch_param_ = param;

    auto ft = this->info_.feature_types.ConstDeviceSpan();
    ellpack_page_source_.reset();  // make sure resource is released before making new ones.
    ellpack_page_source_ = std::make_shared<EllpackPageSource>(
        this->missing_, ctx->Threads(), this->Info().num_col_, this->n_batches_, cache_info_.at(id),
        param, std::move(cuts), this->IsDense(), row_stride, ft, sparse_page_source_, ctx->gpu_id);
  } else {
    CHECK(sparse_page_source_);
    ellpack_page_source_->Reset();
  }

  auto begin_iter = BatchIterator<EllpackPage>(ellpack_page_source_);
  return BatchSet<EllpackPage>(BatchIterator<EllpackPage>(begin_iter));
}
}  // namespace xgboost::data
