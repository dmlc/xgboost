/*!
 * Copyright 2021 by Contributors
 */
#include "array_interface.h"
#include "../common/common.h"

namespace xgboost {
void ArrayInterfaceHandler::SyncCudaStream(ptrdiff_t stream) {
  dh::safe_cuda(hipStreamSynchronize(reinterpret_cast<hipStream_t>(stream)));
}
}  // namespace xgboost
