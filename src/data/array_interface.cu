/**
 * Copyright 2021-2023, XGBoost Contributors
 */
#include <cstdint>  // for int64_t

#include "../common/common.h"
#include "../common/device_helpers.cuh"  // for DefaultStream, CUDAEvent
#include "array_interface.h"
#include "xgboost/logging.h"

namespace xgboost {
void ArrayInterfaceHandler::SyncCudaStream(std::int64_t stream) {
  switch (stream) {
    case 0:
      /**
       * disallowed by the `__cuda_array_interface__`.  Quote:
       *
       *   This is disallowed as it would be ambiguous between None and the default
       *   stream, and also between the legacy and per-thread default streams. Any use
       *   case where 0 might be given should either use None, 1, or 2 instead for
       *   clarity.
       */
      LOG(FATAL) << "Invalid stream ID in array interface: " << stream;
    case 1:
      // default legacy stream
      break;
    case 2:
      // default per-thread stream
    default: {
      dh::CUDAEvent e;
      e.Record(dh::CUDAStreamView{reinterpret_cast<hipStream_t>(stream)});
      dh::DefaultStream().Wait(e);
    }
  }
}

bool ArrayInterfaceHandler::IsCudaPtr(void const* ptr) {
  if (!ptr) {
    return false;
  }
  hipPointerAttribute_t attr;
  auto err = hipPointerGetAttributes(&attr, ptr);
  // reset error
  CHECK_EQ(err, hipGetLastError());
  if (err == hipErrorInvalidValue) {
    // CUDA < 11
    return false;
  } else if (err == hipSuccess) {
    // CUDA >= 11
    switch (attr.type) {
      case cudaMemoryTypeUnregistered:
      case hipMemoryTypeHost:
        return false;
      default:
        return true;
    }
    return true;
  } else {
    // other errors, `hipErrorNoDevice`, `hipErrorInsufficientDriver` etc.
    return false;
  }
}
}  // namespace xgboost
