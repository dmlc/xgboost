/**
 * Copyright 2019-2024, XGBoost contributors
 */
#include <thrust/host_vector.h>  // for host_vector

#include <cstddef>  // for size_t
#include <cstdint>  // for int8_t, uint64_t, uint32_t
#include <memory>   // for shared_ptr, make_unique, make_shared
#include <numeric>  // for accumulate
#include <utility>  // for move

#include "../common/common.h"                 // for safe_cuda
#include "../common/ref_resource_view.cuh"
#include "../common/device_helpers.cuh"       // for CUDAStreamView, DefaultStream
#include "../common/resource.cuh"             // for PrivateCudaMmapConstStream
#include "ellpack_page.cuh"                   // for EllpackPageImpl
#include "ellpack_page.h"                     // for EllpackPage
#include "ellpack_page_source.h"
#include "proxy_dmatrix.cuh"  // for Dispatch
#include "xgboost/base.h"     // for bst_idx_t
#include "../common/transform_iterator.h"  // for MakeIndexTransformIter

namespace xgboost::data {
/**
 * Cache
 */
EllpackHostCache::EllpackHostCache() = default;
EllpackHostCache::~EllpackHostCache() = default;

[[nodiscard]] std::size_t EllpackHostCache::Size() const {
  auto it = common::MakeIndexTransformIter([&](auto i) { return pages.at(i)->MemCostBytes(); });
  return std::accumulate(it, it + pages.size(), 0l);
}

void EllpackHostCache::Push(std::unique_ptr<EllpackPageImpl> page) {
  this->pages.emplace_back(std::move(page));
}

EllpackPageImpl const* EllpackHostCache::Get(std::int32_t k) {
  return this->pages.at(k).get();
}

/**
 * Cache stream.
 */
class EllpackHostCacheStreamImpl {
  std::shared_ptr<EllpackHostCache> cache_;
  std::int32_t ptr_;

 public:
  explicit EllpackHostCacheStreamImpl(std::shared_ptr<EllpackHostCache> cache)
      : cache_{std::move(cache)} {}

  auto Share() { return cache_; }

  void Seek(bst_idx_t offset_bytes) {
    std::size_t n_bytes{0};
    std::int32_t k{-1};
    for (std::size_t i = 0, n = cache_->pages.size(); i < n; ++i) {
      if (n_bytes == offset_bytes) {
        k = i;
        break;
      }
      n_bytes += cache_->pages[i]->MemCostBytes();
    }
    if (offset_bytes == n_bytes && k == -1) {
      k = this->cache_->pages.size();  // seek end
    }
    CHECK_NE(k, -1) << "Invalid offset:" << offset_bytes;
    ptr_ = k;
  }

  void Write(EllpackPage const& page) {
    auto impl = page.Impl();

    auto new_impl = std::make_unique<EllpackPageImpl>();
    auto new_cache = std::make_shared<EllpackHostCache>();
    new_impl->gidx_buffer =
        common::MakeFixedVecWithPinnedMalloc<common::CompressedByteT>(impl->gidx_buffer.size());
    new_impl->n_rows = impl->Size();
    new_impl->is_dense = impl->IsDense();
    new_impl->row_stride = impl->row_stride;
    new_impl->base_rowid = impl->base_rowid;

    dh::safe_cuda(hipMemcpyAsync(new_impl->gidx_buffer.data(), impl->gidx_buffer.data(),
                                  impl->gidx_buffer.size_bytes(), hipMemcpyDefault));

    this->cache_->Push(std::move(new_impl));
    ptr_ += 1;
  }

  void Read(EllpackPage* out, bool prefetch_copy) const {
    auto page = this->cache_->Get(ptr_);

    auto impl = out->Impl();
    if (prefetch_copy) {
      impl->gidx_buffer =
          common::MakeFixedVecWithCudaMalloc<common::CompressedByteT>(page->gidx_buffer.size());
      dh::safe_cuda(hipMemcpyAsync(impl->gidx_buffer.data(), page->gidx_buffer.data(),
                                    page->gidx_buffer.size_bytes(), hipMemcpyDefault));
    } else {
      auto res = page->gidx_buffer.Resource();
      impl->gidx_buffer = common::RefResourceView<common::CompressedByteT>{
          res->DataAs<common::CompressedByteT>(), page->gidx_buffer.size(), res};
    }

    impl->n_rows = page->Size();
    impl->is_dense = page->IsDense();
    impl->row_stride = page->row_stride;
    impl->base_rowid = page->base_rowid;
  }
};

/**
 * EllpackHostCacheStream
 */

EllpackHostCacheStream::EllpackHostCacheStream(std::shared_ptr<EllpackHostCache> cache)
    : p_impl_{std::make_unique<EllpackHostCacheStreamImpl>(std::move(cache))} {}

EllpackHostCacheStream::~EllpackHostCacheStream() = default;

std::shared_ptr<EllpackHostCache> EllpackHostCacheStream::Share() { return p_impl_->Share(); }

void EllpackHostCacheStream::Seek(bst_idx_t offset_bytes) { this->p_impl_->Seek(offset_bytes); }

void EllpackHostCacheStream::Read(EllpackPage* page, bool prefetch_copy) const {
  this->p_impl_->Read(page, prefetch_copy);
}

void EllpackHostCacheStream::Write(EllpackPage const& page) { this->p_impl_->Write(page); }

/**
 * EllpackCacheStreamPolicy
 */

template <typename S, template <typename> typename F>
EllpackCacheStreamPolicy<S, F>::EllpackCacheStreamPolicy()
    : p_cache_{std::make_shared<EllpackHostCache>()} {}

template <typename S, template <typename> typename F>
[[nodiscard]] std::unique_ptr<typename EllpackCacheStreamPolicy<S, F>::WriterT>
EllpackCacheStreamPolicy<S, F>::CreateWriter(StringView, std::uint32_t iter) {
  auto fo = std::make_unique<EllpackHostCacheStream>(this->p_cache_);
  if (iter == 0) {
    CHECK(this->p_cache_->Empty());
  } else {
    fo->Seek(this->p_cache_->Size());
  }
  return fo;
}

template <typename S, template <typename> typename F>
[[nodiscard]] std::unique_ptr<typename EllpackCacheStreamPolicy<S, F>::ReaderT>
EllpackCacheStreamPolicy<S, F>::CreateReader(StringView, bst_idx_t offset, bst_idx_t) const {
  auto fi = std::make_unique<ReaderT>(this->p_cache_);
  fi->Seek(offset);
  return fi;
}

// Instantiation
template EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>::EllpackCacheStreamPolicy();

template std::unique_ptr<
    typename EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>::WriterT>
EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>::CreateWriter(StringView name,
                                                                         std::uint32_t iter);

template std::unique_ptr<
    typename EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>::ReaderT>
EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>::CreateReader(StringView name,
                                                                         bst_idx_t offset,
                                                                         bst_idx_t length) const;

/**
 * EllpackMmapStreamPolicy
 */

template <typename S, template <typename> typename F>
[[nodiscard]] std::unique_ptr<typename EllpackMmapStreamPolicy<S, F>::ReaderT>
EllpackMmapStreamPolicy<S, F>::CreateReader(StringView name, bst_idx_t offset,
                                            bst_idx_t length) const {
  if (has_hmm_) {
    return std::make_unique<common::PrivateCudaMmapConstStream>(name, offset, length);
  } else {
    return std::make_unique<common::PrivateMmapConstStream>(name, offset, length);
  }
}

// Instantiation
template std::unique_ptr<
    typename EllpackMmapStreamPolicy<EllpackPage, EllpackFormatPolicy>::ReaderT>
EllpackMmapStreamPolicy<EllpackPage, EllpackFormatPolicy>::CreateReader(StringView name,
                                                                        bst_idx_t offset,
                                                                        bst_idx_t length) const;

/**
 * EllpackPageSourceImpl
 */
template <typename F>
void EllpackPageSourceImpl<F>::Fetch() {
  dh::safe_cuda(hipSetDevice(this->Device().ordinal));
  if (!this->ReadCache()) {
    if (this->count_ != 0 && !this->sync_) {
      // source is initialized to be the 0th page during construction, so when count_ is 0
      // there's no need to increment the source.
      ++(*this->source_);
    }
    // This is not read from cache so we still need it to be synced with sparse page source.
    CHECK_EQ(this->count_, this->source_->Iter());
    auto const& csr = this->source_->Page();
    this->page_.reset(new EllpackPage{});
    auto* impl = this->page_->Impl();
    Context ctx = Context{}.MakeCUDA(this->Device().ordinal);
    *impl = EllpackPageImpl{&ctx, this->GetCuts(), *csr, is_dense_, row_stride_, feature_types_};
    this->page_->SetBaseRowId(csr->base_rowid);
    LOG(INFO) << "Generated an Ellpack page with size: " << impl->MemCostBytes()
              << " from a SparsePage with size:" << csr->MemCostBytes();
    this->WriteCache();
  }
}

// Instantiation
template void
EllpackPageSourceImpl<DefaultFormatStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
template void
EllpackPageSourceImpl<EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
template void
EllpackPageSourceImpl<EllpackMmapStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();

/**
 * ExtEllpackPageSourceImpl
 */
template <typename F>
void ExtEllpackPageSourceImpl<F>::Fetch() {
  dh::safe_cuda(hipSetDevice(this->Device().ordinal));
  if (!this->ReadCache()) {
    auto iter = this->source_->Iter();
    CHECK_EQ(this->count_, iter);
    ++(*this->source_);
    CHECK_GE(this->source_->Iter(), 1);
    cuda_impl::Dispatch(proxy_, [this](auto const& value) {
      CHECK(this->proxy_->Ctx()->IsCUDA()) << "All batches must use the same device type.";
      proxy_->Info().feature_types.SetDevice(dh::GetDevice(this->ctx_));
      auto d_feature_types = proxy_->Info().feature_types.ConstDeviceSpan();
      auto n_samples = value.NumRows();

      dh::device_vector<size_t> row_counts(n_samples + 1, 0);
      common::Span<size_t> row_counts_span(row_counts.data().get(), row_counts.size());
      cuda_impl::Dispatch(proxy_, [=](auto const& value) {
        return GetRowCounts(value, row_counts_span, dh::GetDevice(this->ctx_), this->missing_);
      });

      this->page_.reset(new EllpackPage{});
      *this->page_->Impl() = EllpackPageImpl{this->ctx_,
                                             value,
                                             this->missing_,
                                             this->info_->IsDense(),
                                             row_counts_span,
                                             d_feature_types,
                                             this->ext_info_.row_stride,
                                             n_samples,
                                             this->GetCuts()};
      this->info_->Extend(proxy_->Info(), false, true);
    });
    this->page_->SetBaseRowId(this->ext_info_.base_rows.at(iter));
    this->WriteCache();
  }
}

// Instantiation
template void
ExtEllpackPageSourceImpl<DefaultFormatStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
template void
ExtEllpackPageSourceImpl<EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
template void
ExtEllpackPageSourceImpl<EllpackMmapStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
}  // namespace xgboost::data
