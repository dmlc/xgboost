#include "hip/hip_runtime.h"
/**
 * Copyright 2019-2025, XGBoost contributors
 */
#include <algorithm>  // for max
#include <cstddef>    // for size_t
#include <cstdint>    // for int8_t, uint64_t, uint32_t
#include <memory>     // for shared_ptr, make_unique, make_shared
#include <numeric>    // for accumulate
#include <utility>    // for move

#include "../common/common.h"               // for HumanMemUnit, safe_cuda
#include "../common/cuda_rt_utils.h"        // for SetDevice
#include "../common/cuda_stream_pool.cuh"   // for StreamPool
#include "../common/device_helpers.cuh"     // for CUDAStreamView, DefaultStream
#include "../common/ref_resource_view.cuh"  // for MakeFixedVecWithCudaMalloc
#include "../common/resource.cuh"           // for PrivateCudaMmapConstStream
#include "../common/transform_iterator.h"   // for MakeIndexTransformIter
#include "batch_utils.h"                    // for HostRatioIsAuto
#include "ellpack_page.cuh"                 // for EllpackPageImpl
#include "ellpack_page.h"                   // for EllpackPage
#include "ellpack_page_source.h"
#include "proxy_dmatrix.cuh"  // for Dispatch
#include "xgboost/base.h"     // for bst_idx_t

namespace xgboost::data {
/**
 * Cache
 */
EllpackMemCache::EllpackMemCache(EllpackCacheInfo cinfo, std::int32_t n_workers)
    : cache_mapping{std::move(cinfo.cache_mapping)},
      buffer_bytes{std::move(cinfo.buffer_bytes)},
      buffer_rows{std::move(cinfo.buffer_rows)},
      cache_host_ratio{cinfo.cache_host_ratio},
      streams{std::make_unique<curt::StreamPool>(n_workers)} {
  CHECK_EQ(buffer_bytes.size(), buffer_rows.size());
  CHECK(!detail::HostRatioIsAuto(this->cache_host_ratio));
  CHECK_GE(this->cache_host_ratio, 0) << error::CacheHostRatioInvalid();
  CHECK_LE(this->cache_host_ratio, 1) << error::CacheHostRatioInvalid();
}

EllpackMemCache::~EllpackMemCache() = default;

[[nodiscard]] std::size_t EllpackMemCache::SizeBytes() const noexcept(true) {
  auto it = common::MakeIndexTransformIter([&](auto i) { return this->SizeBytes(i); });
  using T = std::iterator_traits<decltype(it)>::value_type;
  return std::accumulate(it, it + this->Size(), static_cast<T>(0));
}

[[nodiscard]] std::size_t EllpackMemCache::DeviceSizeBytes() const noexcept(true) {
  auto it =
      common::MakeIndexTransformIter([&](auto i) { return this->d_pages.at(i).size_bytes(); });
  using T = std::iterator_traits<decltype(it)>::value_type;
  return std::accumulate(it, it + this->Size(), static_cast<T>(0));
}

[[nodiscard]] std::size_t EllpackMemCache::SizeBytes(std::size_t i) const noexcept(true) {
  return this->h_pages.at(i)->MemCostBytes() + this->d_pages.at(i).size_bytes();
}

[[nodiscard]] std::size_t EllpackMemCache::GidxSizeBytes(std::size_t i) const noexcept(true) {
  return this->h_pages.at(i)->gidx_buffer.size_bytes() + this->d_pages.at(i).size_bytes();
}

[[nodiscard]] std::size_t EllpackMemCache::GidxSizeBytes() const noexcept(true) {
  auto it = common::MakeIndexTransformIter([&](auto i) { return this->GidxSizeBytes(i); });
  using T = std::iterator_traits<decltype(it)>::value_type;
  return std::accumulate(it, it + this->Size(), static_cast<T>(0));
}

[[nodiscard]] EllpackMemCache::PagePtr EllpackMemCache::At(std::int32_t k) const {
  auto const* h_ptr = this->h_pages.at(k).get();
  auto const* d_ptr = &this->d_pages.at(k);
  return std::make_pair(h_ptr, d_ptr);
}

[[nodiscard]] EllpackMemCache::PageRef EllpackMemCache::Back() {
  auto& h_ref = this->h_pages.back();
  auto& d_ref = this->d_pages.back();
  return {h_ref, d_ref};
}

/**
 * Cache stream.
 */
class EllpackHostCacheStreamImpl {
  std::shared_ptr<EllpackMemCache> cache_;
  std::int32_t ptr_{0};

 public:
  explicit EllpackHostCacheStreamImpl(std::shared_ptr<EllpackMemCache> cache)
      : cache_{std::move(cache)} {}

  auto Share() const { return this->cache_; }

  void Seek(bst_idx_t offset_bytes) {
    std::size_t n_bytes{0};
    std::int32_t k{-1};
    for (std::size_t i = 0, n = cache_->h_pages.size(); i < n; ++i) {
      if (n_bytes == offset_bytes) {
        k = i;
        break;
      }
      n_bytes += this->cache_->SizeBytes(i);
    }
    if (offset_bytes == n_bytes && k == -1) {
      k = this->cache_->h_pages.size();  // seek end
    }
    CHECK_NE(k, -1) << "Invalid offset:" << offset_bytes;
    ptr_ = k;
  }

  [[nodiscard]] bool Write(EllpackPage const& page) {
    auto impl = page.Impl();
    auto ctx = Context{}.MakeCUDA(dh::CurrentDevice());

    this->cache_->sizes_orig.push_back(page.Impl()->MemCostBytes());
    auto orig_ptr = this->cache_->sizes_orig.size() - 1;

    CHECK_LT(orig_ptr, this->cache_->NumBatchesOrig());
    auto cache_idx = this->cache_->cache_mapping.at(orig_ptr);
    // Wrap up the previous page if this is a new page, or this is the last page.
    auto new_page = cache_idx == this->cache_->h_pages.size();
    // Last page expected from the user.
    auto last_page = (orig_ptr + 1) == this->cache_->NumBatchesOrig();

    bool const no_concat = this->cache_->NoConcat();

    auto cache_host_ratio = this->cache_->cache_host_ratio;
    CHECK_GE(cache_host_ratio, 0) << error::CacheHostRatioInvalid();
    CHECK_LE(cache_host_ratio, 1) << error::CacheHostRatioInvalid();

    // Get the size of the host cache.
    auto get_host_nbytes = [&](EllpackPageImpl const* old_impl) {
      if (this->cache_->cache_host_ratio == 1.0) {
        return old_impl->gidx_buffer.size_bytes();
      }
      if (this->cache_->cache_host_ratio == 0.0) {
        return static_cast<std::size_t>(0);
      }
      auto n_bytes =
          std::max(static_cast<std::size_t>(old_impl->gidx_buffer.size_bytes() * cache_host_ratio),
                   std::size_t{1});
      return n_bytes;
    };
    // Finish writing a (concatenated) cache page.
    auto commit_page = [cache_host_ratio, get_host_nbytes](EllpackPageImpl const* old_impl) {
      CHECK_EQ(old_impl->gidx_buffer.Resource()->Type(), common::ResourceHandler::kCudaMalloc);
      auto new_impl = std::make_unique<EllpackPageImpl>();
      new_impl->CopyInfo(old_impl);
      // Split the cache into host cache and device cache

      // Host cache
      auto n_bytes = get_host_nbytes(old_impl);
      CHECK_LE(n_bytes, old_impl->gidx_buffer.size_bytes());
      new_impl->gidx_buffer =
          common::MakeFixedVecWithPinnedMalloc<common::CompressedByteT>(n_bytes);
      if (n_bytes > 0) {
        dh::safe_cuda(hipMemcpyAsync(new_impl->gidx_buffer.data(), old_impl->gidx_buffer.data(),
                                      n_bytes, hipMemcpyDefault));
      }

      // Device cache
      auto remaining = old_impl->gidx_buffer.size_bytes() - n_bytes;
      auto d_page = common::MakeFixedVecWithCudaMalloc<common::CompressedByteT>(remaining);
      if (remaining > 0) {
        dh::safe_cuda(hipMemcpyAsync(d_page.data(), old_impl->gidx_buffer.data() + n_bytes,
                                      remaining, hipMemcpyDefault));
      }
      CHECK_LE(new_impl->gidx_buffer.size(), old_impl->gidx_buffer.size());
      CHECK_EQ(new_impl->MemCostBytes() + d_page.size_bytes(), old_impl->MemCostBytes());
      LOG(INFO) << "Create cache page with size:"
                << common::HumanMemUnit(new_impl->MemCostBytes() + d_page.size_bytes());
      return std::make_pair(std::move(new_impl), std::move(d_page));
    };

    if (no_concat) {
      CHECK(new_page);
      auto old_impl = page.Impl();
      auto [commited, d_page] = commit_page(old_impl);

      this->cache_->offsets.push_back(old_impl->n_rows * old_impl->info.row_stride);
      this->cache_->h_pages.emplace_back(std::move(commited));
      this->cache_->d_pages.emplace_back(std::move(d_page));
      return new_page;
    }

    if (new_page) {
      if (!this->cache_->h_pages.empty()) {
        // Need to wrap up the previous page.
        // Replace the previous page (on device) with a new page on host.
        this->cache_->Back() = commit_page(this->cache_->h_pages.back().get());
      }
      // Push a new page
      auto n_bytes = this->cache_->buffer_bytes.at(this->cache_->h_pages.size());
      auto n_samples = this->cache_->buffer_rows.at(this->cache_->h_pages.size());
      auto new_impl = std::make_unique<EllpackPageImpl>(&ctx, impl->CutsShared(), impl->IsDense(),
                                                        impl->info.row_stride, n_samples);
      new_impl->SetBaseRowId(impl->base_rowid);
      new_impl->SetNumSymbols(impl->NumSymbols());
      new_impl->gidx_buffer =
          common::MakeFixedVecWithCudaMalloc<common::CompressedByteT>(&ctx, n_bytes, 0);
      auto offset = new_impl->Copy(&ctx, impl, 0);

      this->cache_->offsets.push_back(offset);

      // Make sure we can always access the back of the vectors
      this->cache_->h_pages.emplace_back(std::move(new_impl));
      this->cache_->d_pages.emplace_back();
    } else {
      // Concatenate into the device pages even though `d_pages` is used. We split the
      // page at the commit stage.
      CHECK(!this->cache_->h_pages.empty());
      CHECK_EQ(cache_idx, this->cache_->h_pages.size() - 1);
      auto& new_impl = this->cache_->h_pages.back();
      auto offset = new_impl->Copy(&ctx, impl, this->cache_->offsets.back());
      this->cache_->offsets.back() += offset;
    }

    // No need to copy if it's already in device.
    if (last_page) {
      this->cache_->Back() = commit_page(this->cache_->h_pages.back().get());
    }

    CHECK_EQ(this->cache_->h_pages.size(), this->cache_->d_pages.size());
    return new_page;
  }

  void Read(EllpackPage* out, bool prefetch_copy) const {
    CHECK_EQ(this->cache_->h_pages.size(), this->cache_->d_pages.size());
    auto [h_page, d_page] = this->cache_->At(this->ptr_);

    auto ctx = Context{}.MakeCUDA(dh::CurrentDevice());
    // FIXME(jiamingy): Accessing split cache directly is not yet supported.
    if (0.0 < this->cache_->cache_host_ratio && this->cache_->cache_host_ratio < 1.0) {
      prefetch_copy = true;
    }
    auto out_impl = out->Impl();
    if (prefetch_copy) {
      auto n_bytes = this->cache_->GidxSizeBytes(this->ptr_);
      out_impl->gidx_buffer = common::MakeFixedVecWithCudaMalloc<common::CompressedByteT>(n_bytes);
      if (!h_page->gidx_buffer.empty()) {
        dh::safe_cuda(hipMemcpyAsync(out_impl->gidx_buffer.data(), h_page->gidx_buffer.data(),
                                      h_page->gidx_buffer.size_bytes(), hipMemcpyDefault,
                                      ctx.CUDACtx()->Stream()));
      }
      if (!d_page->empty()) {
        auto beg = out_impl->gidx_buffer.data() + h_page->gidx_buffer.size();
        dh::safe_cuda(hipMemcpyAsync(beg, d_page->data(), d_page->size_bytes(), hipMemcpyDefault,
                                      ctx.CUDACtx()->Stream()));
      }
    } else {
      CHECK(d_page->empty() || h_page->gidx_buffer.empty());
      auto res = d_page->empty() ? h_page->gidx_buffer.Resource() : d_page->Resource();
      out_impl->gidx_buffer = common::RefResourceView<common::CompressedByteT>{
          res->DataAs<common::CompressedByteT>(), h_page->gidx_buffer.size(), res};
    }

    out_impl->CopyInfo(h_page);
  }
};

/**
 * EllpackHostCacheStream
 */
EllpackHostCacheStream::EllpackHostCacheStream(std::shared_ptr<EllpackMemCache> cache)
    : p_impl_{std::make_unique<EllpackHostCacheStreamImpl>(std::move(cache))} {}

EllpackHostCacheStream::~EllpackHostCacheStream() = default;

std::shared_ptr<EllpackMemCache const> EllpackHostCacheStream::Share() const {
  return p_impl_->Share();
}

void EllpackHostCacheStream::Seek(bst_idx_t offset_bytes) { this->p_impl_->Seek(offset_bytes); }

void EllpackHostCacheStream::Read(EllpackPage* page, bool prefetch_copy) const {
  this->p_impl_->Read(page, prefetch_copy);
}

[[nodiscard]] bool EllpackHostCacheStream::Write(EllpackPage const& page) {
  return this->p_impl_->Write(page);
}

/**
 * EllpackCacheStreamPolicy
 */
template <typename S, template <typename> typename F>
[[nodiscard]] std::unique_ptr<typename EllpackCacheStreamPolicy<S, F>::WriterT>
EllpackCacheStreamPolicy<S, F>::CreateWriter(StringView, std::uint32_t iter) {
  if (!this->p_cache_) {
    CHECK(!detail::HostRatioIsAuto(this->CacheInfo().cache_host_ratio));
    CHECK_GE(this->CacheInfo().cache_host_ratio, 0.0);
    CHECK_LE(this->CacheInfo().cache_host_ratio, 1.0);
    constexpr std::int32_t kMaxGpuExtMemWorkers = 4;
    this->p_cache_ = std::make_unique<EllpackMemCache>(this->CacheInfo(), kMaxGpuExtMemWorkers);
  }
  auto fo = std::make_unique<EllpackHostCacheStream>(this->p_cache_);
  if (iter == 0) {
    CHECK(this->p_cache_->Empty());
  } else {
    fo->Seek(this->p_cache_->SizeBytes());
  }
  return fo;
}

template <typename S, template <typename> typename F>
[[nodiscard]] std::unique_ptr<typename EllpackCacheStreamPolicy<S, F>::ReaderT>
EllpackCacheStreamPolicy<S, F>::CreateReader(StringView, bst_idx_t offset, bst_idx_t) const {
  auto fi = std::make_unique<ReaderT>(this->p_cache_);
  fi->Seek(offset);
  return fi;
}

// Instantiation
template std::unique_ptr<
    typename EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>::WriterT>
EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>::CreateWriter(StringView name,
                                                                         std::uint32_t iter);

template std::unique_ptr<
    typename EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>::ReaderT>
EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>::CreateReader(StringView name,
                                                                         bst_idx_t offset,
                                                                         bst_idx_t length) const;

/**
 * EllpackMmapStreamPolicy
 */

template <typename S, template <typename> typename F>
[[nodiscard]] std::unique_ptr<typename EllpackMmapStreamPolicy<S, F>::ReaderT>
EllpackMmapStreamPolicy<S, F>::CreateReader(StringView name, bst_idx_t offset,
                                            bst_idx_t length) const {
  if (has_hmm_) {
    return std::make_unique<common::PrivateCudaMmapConstStream>(name, offset, length);
  } else {
    return std::make_unique<common::PrivateMmapConstStream>(name, offset, length);
  }
}

// Instantiation
template std::unique_ptr<
    typename EllpackMmapStreamPolicy<EllpackPage, EllpackFormatPolicy>::ReaderT>
EllpackMmapStreamPolicy<EllpackPage, EllpackFormatPolicy>::CreateReader(StringView name,
                                                                        bst_idx_t offset,
                                                                        bst_idx_t length) const;

void CalcCacheMapping(Context const* ctx, bool is_dense,
                      std::shared_ptr<common::HistogramCuts const> cuts,
                      std::int64_t min_cache_page_bytes, ExternalDataInfo const& ext_info,
                      EllpackCacheInfo* cinfo) {
  CHECK(cinfo->param.Initialized()) << "Need to initialize scalar fields first.";
  auto ell_info = CalcNumSymbols(ctx, ext_info.row_stride, is_dense, cuts);
  std::vector<std::size_t> cache_bytes;
  std::vector<std::size_t> cache_mapping(ext_info.n_batches, 0);
  std::vector<std::size_t> cache_rows;

  for (std::size_t i = 0; i < ext_info.n_batches; ++i) {
    auto n_samples = ext_info.base_rowids.at(i + 1) - ext_info.base_rowids[i];
    auto n_bytes = common::CompressedBufferWriter::CalculateBufferSize(
        ext_info.row_stride * n_samples, ell_info.n_symbols);
    if (cache_bytes.empty()) {
      // Push the first page
      cache_bytes.push_back(n_bytes);
      cache_rows.push_back(n_samples);
    } else if (static_cast<decltype(min_cache_page_bytes)>(cache_bytes.back()) <
               min_cache_page_bytes) {
      // Concatenate to the previous page
      cache_bytes.back() += n_bytes;
      cache_rows.back() += n_samples;
    } else {
      // Push a new page
      cache_bytes.push_back(n_bytes);
      cache_rows.push_back(n_samples);
    }
    cache_mapping[i] = cache_bytes.size() - 1;
  }

  cinfo->cache_mapping = std::move(cache_mapping);
  cinfo->buffer_bytes = std::move(cache_bytes);
  cinfo->buffer_rows = std::move(cache_rows);
  // Directly store in device if there's only one batch.
  if (cinfo->NumBatchesCc() == 1) {
    cinfo->cache_host_ratio = 0.0;
    LOG(INFO) << "Prefer device cache as there's only 1 page.";
  }
}

/**
 * EllpackPageSourceImpl
 */
template <typename F>
void EllpackPageSourceImpl<F>::Fetch() {
  curt::SetDevice(this->Device().ordinal);
  if (!this->ReadCache()) {
    if (this->Iter() != 0 && !this->sync_) {
      // source is initialized to be the 0th page during construction, so when count_ is 0
      // there's no need to increment the source.
      ++(*this->source_);
    }
    // This is not read from cache so we still need it to be synced with sparse page source.
    CHECK_EQ(this->Iter(), this->source_->Iter());
    auto const& csr = this->source_->Page();
    this->page_.reset(new EllpackPage{});
    auto* impl = this->page_->Impl();
    Context ctx = Context{}.MakeCUDA(this->Device().ordinal);
    if (this->GetCuts()->HasCategorical()) {
      CHECK(!this->feature_types_.empty());
    }
    *impl = EllpackPageImpl{&ctx, this->GetCuts(), *csr, is_dense_, row_stride_, feature_types_};
    this->page_->SetBaseRowId(csr->base_rowid);
    LOG(INFO) << "Generated an Ellpack page with size: "
              << common::HumanMemUnit(impl->MemCostBytes())
              << " from a SparsePage with size:" << common::HumanMemUnit(csr->MemCostBytes());
    this->WriteCache();
  }
}

// Instantiation
template void
EllpackPageSourceImpl<DefaultFormatStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
template void
EllpackPageSourceImpl<EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
template void
EllpackPageSourceImpl<EllpackMmapStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();

/**
 * ExtEllpackPageSourceImpl
 */
template <typename F>
void ExtEllpackPageSourceImpl<F>::Fetch() {
  curt::SetDevice(this->Device().ordinal);
  if (!this->ReadCache()) {
    auto iter = this->source_->Iter();
    CHECK_EQ(this->Iter(), iter);
    cuda_impl::Dispatch(proxy_, [this](auto const& value) {
      CHECK(this->proxy_->Ctx()->IsCUDA()) << "All batches must use the same device type.";
      proxy_->Info().feature_types.SetDevice(dh::GetDevice(this->ctx_));
      auto d_feature_types = proxy_->Info().feature_types.ConstDeviceSpan();
      auto n_samples = value.NumRows();
      if (this->GetCuts()->HasCategorical()) {
        CHECK(!d_feature_types.empty());
      }
      dh::device_vector<size_t> row_counts(n_samples + 1, 0);
      common::Span<size_t> row_counts_span(row_counts.data().get(), row_counts.size());
      bst_idx_t row_stride = GetRowCounts(this->ctx_, value, row_counts_span,
                                          dh::GetDevice(this->ctx_), this->missing_);
      CHECK_LE(row_stride, this->ext_info_.row_stride);
      this->page_.reset(new EllpackPage{});
      *this->page_->Impl() = EllpackPageImpl{this->ctx_,
                                             value,
                                             this->missing_,
                                             this->info_->IsDense(),
                                             row_counts_span,
                                             d_feature_types,
                                             this->ext_info_.row_stride,
                                             n_samples,
                                             this->GetCuts()};
      this->info_->Extend(proxy_->Info(), false, true);
    });
    LOG(INFO) << "Generated an Ellpack page with size: "
              << common::HumanMemUnit(this->page_->Impl()->MemCostBytes())
              << " from an batch with estimated size: "
              << cuda_impl::Dispatch<false>(proxy_, [](auto const& adapter) {
                   return common::HumanMemUnit(adapter->SizeBytes());
                 });
    this->page_->SetBaseRowId(this->ext_info_.base_rowids.at(iter));
    this->WriteCache();
  }
}

// Instantiation
template void
ExtEllpackPageSourceImpl<DefaultFormatStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
template void
ExtEllpackPageSourceImpl<EllpackCacheStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
template void
ExtEllpackPageSourceImpl<EllpackMmapStreamPolicy<EllpackPage, EllpackFormatPolicy>>::Fetch();
}  // namespace xgboost::data
