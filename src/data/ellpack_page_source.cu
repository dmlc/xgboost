/*!
 * Copyright 2019 XGBoost contributors
 */
#include <memory>
#include <utility>
#include <vector>

#include "../common/hist_util.h"

#include "ellpack_page_source.h"
#include "sparse_page_source.h"
#include "ellpack_page.cuh"

namespace xgboost {
namespace data {

class EllpackPageSourceImpl : public DataSource<EllpackPage> {
 public:
  /*!
   * \brief Create source from cache files the cache_prefix.
   * \param cache_prefix The prefix of cache we want to solve.
   */
  explicit EllpackPageSourceImpl(DMatrix* dmat,
                                 const std::string& cache_info,
                                 const BatchParam& param) noexcept(false);

  /*! \brief destructor */
  ~EllpackPageSourceImpl() override = default;

  void BeforeFirst() override;
  bool Next() override;
  EllpackPage& Value();
  const EllpackPage& Value() const override;

 private:
  /*! \brief Write Ellpack pages after accumulating them in memory. */
  void WriteEllpackPages(DMatrix* dmat, const std::string& cache_info) const;

  /*! \brief The page type string for ELLPACK. */
  const std::string kPageType_{".ellpack.page"};

  int device_{-1};
  size_t page_size_{DMatrix::kPageSize};
  common::Monitor monitor_;
  dh::BulkAllocator ba_;
  /*! \brief The EllpackInfo, with the underlying GPU memory shared by all pages. */
  EllpackInfo ellpack_info_;
  std::unique_ptr<SparsePageSource<EllpackPage>> source_;
  std::string cache_info_;
};

EllpackPageSource::EllpackPageSource(DMatrix* dmat,
                                     const std::string& cache_info,
                                     const BatchParam& param) noexcept(false)
    : impl_{new EllpackPageSourceImpl(dmat, cache_info, param)} {}

void EllpackPageSource::BeforeFirst() {
  impl_->BeforeFirst();
}

bool EllpackPageSource::Next() {
  return impl_->Next();
}

EllpackPage& EllpackPageSource::Value() {
  return impl_->Value();
}

const EllpackPage& EllpackPageSource::Value() const {
  return impl_->Value();
}

// Build the quantile sketch across the whole input data, then use the histogram cuts to compress
// each CSR page, and write the accumulated ELLPACK pages to disk.
EllpackPageSourceImpl::EllpackPageSourceImpl(DMatrix* dmat,
                                             const std::string& cache_info,
                                             const BatchParam& param) noexcept(false)
    : device_(param.gpu_id), cache_info_(cache_info) {

  if (param.gpu_page_size > 0) {
    page_size_ = param.gpu_page_size;
  }

  monitor_.Init("ellpack_page_source");
  dh::safe_cuda(hipSetDevice(device_));

  monitor_.StartCuda("Quantiles");
  common::HistogramCuts hmat;
  size_t row_stride =
      common::DeviceSketch(device_, param.max_bin, param.gpu_batch_nrows, dmat, &hmat);
  monitor_.StopCuda("Quantiles");

  monitor_.StartCuda("CreateEllpackInfo");
  ellpack_info_ = EllpackInfo(device_, dmat->IsDense(), row_stride, hmat, &ba_);
  monitor_.StopCuda("CreateEllpackInfo");

  monitor_.StartCuda("WriteEllpackPages");
  WriteEllpackPages(dmat, cache_info);
  monitor_.StopCuda("WriteEllpackPages");

  source_.reset(new SparsePageSource<EllpackPage>(cache_info_, kPageType_));
}

void EllpackPageSourceImpl::BeforeFirst() {
  source_.reset(new SparsePageSource<EllpackPage>(cache_info_, kPageType_));
  source_->BeforeFirst();
}

bool EllpackPageSourceImpl::Next() {
  return source_->Next();
}

EllpackPage& EllpackPageSourceImpl::Value() {
  EllpackPage& page = source_->Value();
  page.Impl()->InitDevice(device_, ellpack_info_);
  return page;
}

const EllpackPage& EllpackPageSourceImpl::Value() const {
  EllpackPage& page = source_->Value();
  page.Impl()->InitDevice(device_, ellpack_info_);
  return page;
}

// Compress each CSR page to ELLPACK, and write the accumulated pages to disk.
void EllpackPageSourceImpl::WriteEllpackPages(DMatrix* dmat, const std::string& cache_info) const {
  auto cinfo = ParseCacheInfo(cache_info, kPageType_);
  const size_t extra_buffer_capacity = 6;
  SparsePageWriter<EllpackPage> writer(
      cinfo.name_shards, cinfo.format_shards, extra_buffer_capacity);
  std::shared_ptr<EllpackPage> page;
  writer.Alloc(&page);
  auto* impl = page->Impl();
  impl->matrix.info = ellpack_info_;
  impl->Clear();

  const MetaInfo& info = dmat->Info();
  size_t bytes_write = 0;
  double tstart = dmlc::GetTime();
  for (const auto& batch : dmat->GetBatches<SparsePage>()) {
    impl->Push(device_, batch);

    size_t mem_cost_bytes = impl->MemCostBytes();
    if (mem_cost_bytes >= page_size_) {
      bytes_write += mem_cost_bytes;
      impl->CompressSparsePage(device_);
      writer.PushWrite(std::move(page));
      writer.Alloc(&page);
      impl = page->Impl();
      impl->matrix.info = ellpack_info_;
      impl->Clear();
      double tdiff = dmlc::GetTime() - tstart;
      LOG(INFO) << "Writing " << kPageType_ << " to " << cache_info << " in "
                << ((bytes_write >> 20UL) / tdiff) << " MB/s, "
                << (bytes_write >> 20UL) << " written";
    }
  }
  if (impl->Size() != 0) {
    impl->CompressSparsePage(device_);
    writer.PushWrite(std::move(page));
  }
}

}  // namespace data
}  // namespace xgboost
