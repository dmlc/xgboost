/*!
 * Copyright 2022 by XGBoost Contributors
 */
#include <memory>  // std::unique_ptr

#include "../common/column_matrix.h"
#include "../common/hist_util.h"  // Index
#include "ellpack_page.cuh"
#include "gradient_index.h"
#include "xgboost/data.h"

namespace xgboost {
// Similar to GHistIndexMatrix::SetIndexData, but without the need for adaptor or bin
// searching. Is there a way to unify the code?
template <typename BinT, typename CompressOffset>
void SetIndexData(Context const* ctx, EllpackPageImpl const* page,
                  std::vector<size_t>* p_hit_count_tloc, CompressOffset&& get_offset,
                  GHistIndexMatrix* out) {
  auto accessor = page->GetHostAccessor();
  auto const kNull = static_cast<bst_bin_t>(accessor.NullValue());

  common::Span<BinT> index_data_span = {out->index.data<BinT>(), out->index.Size()};
  auto n_bins_total = page->Cuts().TotalBins();

  auto& hit_count_tloc = *p_hit_count_tloc;
  hit_count_tloc.clear();
  hit_count_tloc.resize(ctx->Threads() * n_bins_total, 0);

  common::ParallelFor(page->Size(), ctx->Threads(), [&](auto i) {
    auto tid = omp_get_thread_num();
    size_t in_rbegin = page->row_stride * i;
    size_t out_rbegin = out->row_ptr[i];
    auto r_size = out->row_ptr[i + 1] - out->row_ptr[i];
    for (size_t j = 0; j < r_size; ++j) {
      auto bin_idx = accessor.gidx_iter[in_rbegin + j];
      assert(bin_idx != kNull);
      index_data_span[out_rbegin + j] = get_offset(bin_idx, j);
      ++hit_count_tloc[tid * n_bins_total + bin_idx];
    }
  });
}

void GetRowPtrFromEllpack(Context const* ctx, EllpackPageImpl const* page,
                          std::vector<size_t>* p_out) {
  auto& row_ptr = *p_out;
  row_ptr.resize(page->Size() + 1, 0);
  if (page->is_dense) {
    std::fill(row_ptr.begin() + 1, row_ptr.end(), page->row_stride);
  } else {
    auto accessor = page->GetHostAccessor();
    auto const kNull = static_cast<bst_bin_t>(accessor.NullValue());

    common::ParallelFor(page->Size(), ctx->Threads(), [&](auto i) {
      size_t ibegin = page->row_stride * i;
      for (size_t j = 0; j < page->row_stride; ++j) {
        bst_bin_t bin_idx = accessor.gidx_iter[ibegin + j];
        if (bin_idx != kNull) {
          row_ptr[i + 1]++;
        }
      }
    });
  }
  std::partial_sum(row_ptr.begin(), row_ptr.end(), row_ptr.begin());
}

GHistIndexMatrix::GHistIndexMatrix(Context const* ctx, MetaInfo const& info,
                                   EllpackPage const& in_page, BatchParam const& p)
    : max_num_bins{p.max_bin} {
  auto page = in_page.Impl();
  isDense_ = page->is_dense;

  CHECK_EQ(info.num_row_, in_page.Size());

  this->cut = page->Cuts();
  // pull to host early, prevent race condition
  this->cut.Ptrs();
  this->cut.Values();
  this->cut.MinValues();

  this->ResizeIndex(info.num_nonzero_, page->is_dense);
  if (page->is_dense) {
    this->index.SetBinOffset(page->Cuts().Ptrs());
  }

  auto n_bins_total = page->Cuts().TotalBins();
  GetRowPtrFromEllpack(ctx, page, &this->row_ptr);
  if (page->is_dense) {
    common::DispatchBinType(this->index.GetBinTypeSize(), [&](auto dtype) {
      using T = decltype(dtype);
      ::xgboost::SetIndexData<T>(ctx, page, &hit_count_tloc_, index.MakeCompressor<T>(), this);
    });
  } else {
    // no compression
    ::xgboost::SetIndexData<uint32_t>(
        ctx, page, &hit_count_tloc_, [&](auto bin_idx, auto) { return bin_idx; }, this);
  }

  this->hit_count.resize(n_bins_total, 0);
  this->GatherHitCount(ctx->Threads(), n_bins_total);

  // sanity checks
  CHECK_EQ(this->Features(), info.num_col_);
  CHECK_EQ(this->Size(), info.num_row_);
  CHECK(this->cut.cut_ptrs_.HostCanRead());
  CHECK(this->cut.cut_values_.HostCanRead());
  CHECK(this->cut.min_vals_.HostCanRead());

  this->columns_ = std::make_unique<common::ColumnMatrix>(*this, p.sparse_thresh);
  this->columns_->InitFromGHist(ctx, *this);
}
}  // namespace xgboost
