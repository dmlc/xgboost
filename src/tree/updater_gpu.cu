#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2018 XGBoost contributors
 */
#include <xgboost/tree_updater.h>
#include <utility>
#include <vector>
#include <limits>
#include <string>

#include "../common/common.h"
#include "param.h"
#include "updater_gpu_common.cuh"

namespace xgboost {
namespace tree {

DMLC_REGISTRY_FILE_TAG(updater_gpu);

template <typename GradientPairT>
XGBOOST_DEVICE float inline LossChangeMissing(const GradientPairT& scan,
                                              const GradientPairT& missing,
                                              const GradientPairT& parent_sum,
                                              const float& parent_gain,
                                              const GPUTrainingParam& param,
                                              bool& missing_left_out) {  // NOLINT
  // Put gradients of missing values to left
  float missing_left_loss =
      DeviceCalcLossChange(param, scan + missing, parent_sum, parent_gain);
  float missing_right_loss =
      DeviceCalcLossChange(param, scan, parent_sum, parent_gain);

  if (missing_left_loss >= missing_right_loss) {
    missing_left_out = true;
    return missing_left_loss;
  } else {
    missing_left_out = false;
    return missing_right_loss;
  }
}

/**
 * @brief Absolute BFS order IDs to col-wise unique IDs based on user input
 * @param tid the index of the element that this thread should access
 * @param abs the array of absolute IDs
 * @param colIds the array of column IDs for each element
 * @param nodeStart the start of the node ID at this level
 * @param nKeys number of nodes at this level.
 * @return the uniq key
 */
static HOST_DEV_INLINE NodeIdT Abs2UniqueKey(int tid,
                                             common::Span<const NodeIdT> abs,
                                             common::Span<const int> colIds,
                                             NodeIdT nodeStart, int nKeys) {
  int a = abs[tid];
  if (a == kUnusedNode) return a;
  return ((a - nodeStart) + (colIds[tid] * nKeys));
}

/**
 * @struct Pair
 * @brief Pair used for key basd scan operations on GradientPair
 */
struct Pair {
  int key;
  GradientPair value;
};

/** define a key that's not used at all in the entire boosting process */
static const int kNoneKey = -100;

/**
 * @brief Allocate temporary buffers needed for scan operations
 * @param tmpScans gradient buffer
 * @param tmpKeys keys buffer
 * @param size number of elements that will be scanned
 */
template <int BLKDIM_L1L3 = 256>
int ScanTempBufferSize(int size) {
  int num_blocks = dh::DivRoundUp(size, BLKDIM_L1L3);
  return num_blocks;
}

struct AddByKey {
  template <typename T>
  HOST_DEV_INLINE T operator()(const T& first, const T& second) const {
    T result;
    if (first.key == second.key) {
      result.key = first.key;
      result.value = first.value + second.value;
    } else {
      result.key = second.key;
      result.value = second.value;
    }
    return result;
  }
};

/**
 * @brief Gradient value getter function
 * @param id the index into the vals or instIds array to which to fetch
 * @param vals the gradient value buffer
 * @param instIds instance index buffer
 * @return the expected gradient value
 */
HOST_DEV_INLINE GradientPair Get(int id,
                                 common::Span<const GradientPair> vals,
                                 common::Span<const int> instIds) {
  id = instIds[id];
  return vals[id];
}

template <int BLKDIM_L1L3>
__global__ void CubScanByKeyL1(
    common::Span<GradientPair> scans,
    common::Span<const GradientPair> vals,
    common::Span<const int> instIds,
    common::Span<GradientPair> mScans,
    common::Span<int> mKeys,
    common::Span<const NodeIdT> keys,
    int nUniqKeys,
    common::Span<const int> colIds, NodeIdT nodeStart,
    const int size) {
  Pair rootPair = {kNoneKey, GradientPair(0.f, 0.f)};
  int myKey;
  GradientPair myValue;
  using BlockScan = hipcub::BlockScan<Pair, BLKDIM_L1L3>;
  __shared__ typename BlockScan::TempStorage temp_storage;
  Pair threadData;
  int tid = blockIdx.x * BLKDIM_L1L3 + threadIdx.x;
  if (tid < size) {
    myKey = Abs2UniqueKey(tid, keys, colIds, nodeStart, nUniqKeys);
    myValue = Get(tid, vals, instIds);
  } else {
    myKey = kNoneKey;
    myValue = {};
  }
  threadData.key = myKey;
  threadData.value = myValue;
  // get previous key, especially needed for the last thread in this block
  // in order to pass on the partial scan values.
  // this statement MUST appear before the checks below!
  // else, the result of this shuffle operation will be undefined
#if (__CUDACC_VER_MAJOR__ >= 9)
  int previousKey = __shfl_up_sync(0xFFFFFFFF, myKey, 1);
#else
  int previousKey = __shfl_up(myKey, 1);
#endif
  // Collectively compute the block-wide exclusive prefix sum
  BlockScan(temp_storage)
      .ExclusiveScan(threadData, threadData, rootPair, AddByKey());
  if (tid < size) {
    scans[tid] = threadData.value;
  } else {
    return;
  }
  if (threadIdx.x == BLKDIM_L1L3 - 1) {
    threadData.value =
        (myKey == previousKey) ? threadData.value : GradientPair(0.0f, 0.0f);
    mKeys[blockIdx.x] = myKey;
    mScans[blockIdx.x] = threadData.value + myValue;
  }
}

template <int BLKSIZE>
__global__ void CubScanByKeyL2(common::Span<GradientPair> mScans,
                               common::Span<int> mKeys, int mLength) {
  using BlockScan = hipcub::BlockScan<Pair, BLKSIZE, hipcub::BLOCK_SCAN_WARP_SCANS>;
  Pair threadData;
  __shared__ typename BlockScan::TempStorage temp_storage;
  for (int i = threadIdx.x; i < mLength; i += BLKSIZE - 1) {
    threadData.key = mKeys[i];
    threadData.value = mScans[i];
    BlockScan(temp_storage).InclusiveScan(threadData, threadData, AddByKey());
    mScans[i] = threadData.value;
    __syncthreads();
  }
}

template <int BLKDIM_L1L3>
__global__ void CubScanByKeyL3(common::Span<GradientPair> sums,
                               common::Span<GradientPair> scans,
                               common::Span<const GradientPair> vals,
                               common::Span<const int> instIds,
                               common::Span<const GradientPair> mScans,
                               common::Span<const int> mKeys,
                               common::Span<const NodeIdT> keys,
                               int nUniqKeys,
                               common::Span<const int> colIds, NodeIdT nodeStart,
                               const int size) {
  int relId = threadIdx.x;
  int tid = (blockIdx.x * BLKDIM_L1L3) + relId;
  // to avoid the following warning from nvcc:
  //   __shared__ memory variable with non-empty constructor or destructor
  //     (potential race between threads)
  __shared__ char gradBuff[sizeof(GradientPair)];
  __shared__ int s_mKeys;
  GradientPair* s_mScans = reinterpret_cast<GradientPair*>(gradBuff);
  if (tid >= size) return;
  // cache block-wide partial scan info
  if (relId == 0) {
    s_mKeys = (blockIdx.x > 0) ? mKeys[blockIdx.x - 1] : kNoneKey;
    s_mScans[0] = (blockIdx.x > 0) ? mScans[blockIdx.x - 1] : GradientPair();
  }
  int myKey = Abs2UniqueKey(tid, keys, colIds, nodeStart, nUniqKeys);
  int previousKey =
      tid == 0 ? kNoneKey
               : Abs2UniqueKey(tid - 1, keys, colIds, nodeStart, nUniqKeys);
  GradientPair my_value = scans[tid];
  __syncthreads();
  if (blockIdx.x > 0 && s_mKeys == previousKey) {
    my_value += s_mScans[0];
  }
  if (tid == size - 1) {
    sums[previousKey] = my_value + Get(tid, vals, instIds);
  }
  if ((previousKey != myKey) && (previousKey >= 0)) {
    sums[previousKey] = my_value;
    my_value = GradientPair(0.0f, 0.0f);
  }
  scans[tid] = my_value;
}

/**
 * @brief Performs fused reduce and scan by key functionality. It is assumed
 * that
 *  the keys occur contiguously!
 * @param sums the output gradient reductions for each element performed
 * key-wise
 * @param scans the output gradient scans for each element performed key-wise
 * @param vals the gradients evaluated for each observation.
 * @param instIds instance ids for each element
 * @param keys keys to be used to segment the reductions. They need not occur
 *  contiguously in contrast to scan_by_key. Currently, we need one key per
 *  value in the 'vals' array.
 * @param size number of elements in the 'vals' array
 * @param nUniqKeys max number of uniq keys found per column
 * @param nCols number of columns
 * @param tmpScans temporary scan buffer needed for cub-pyramid algo
 * @param tmpKeys temporary key buffer needed for cub-pyramid algo
 * @param colIds column indices for each element in the array
 * @param nodeStart index of the leftmost node in the current level
 */
template <int BLKDIM_L1L3 = 256, int BLKDIM_L2 = 512>
void ReduceScanByKey(common::Span<GradientPair> sums,
                     common::Span<GradientPair> scans,
                     common::Span<GradientPair> vals,
                     common::Span<const int> instIds,
                     common::Span<const NodeIdT> keys,
                     int size, int nUniqKeys, int nCols,
                     common::Span<GradientPair> tmpScans,
                     common::Span<int> tmpKeys,
                     common::Span<const int> colIds, NodeIdT nodeStart) {
  int nBlks = dh::DivRoundUp(size, BLKDIM_L1L3);
  hipMemset(sums.data(), 0, nUniqKeys * nCols * sizeof(GradientPair));
  CubScanByKeyL1<BLKDIM_L1L3>
      <<<nBlks, BLKDIM_L1L3>>>(scans, vals, instIds, tmpScans, tmpKeys, keys,
                               nUniqKeys, colIds, nodeStart, size);
  CubScanByKeyL2<BLKDIM_L2><<<1, BLKDIM_L2>>>(tmpScans, tmpKeys, nBlks);
  CubScanByKeyL3<BLKDIM_L1L3>
      <<<nBlks, BLKDIM_L1L3>>>(sums, scans, vals, instIds, tmpScans, tmpKeys,
                               keys, nUniqKeys, colIds, nodeStart, size);
}

/**
 * @struct ExactSplitCandidate
 * @brief Abstraction of a possible split in the decision tree
 */
struct ExactSplitCandidate {
  /** the optimal gain score for this node */
  float score;
  /** index where to split in the DMatrix */
  int index;

  HOST_DEV_INLINE ExactSplitCandidate() : score{-FLT_MAX}, index{INT_MAX} {}

  /**
   * @brief Whether the split info is valid to be used to create a new child
   * @param minSplitLoss minimum score above which decision to split is made
   * @return true if splittable, else false
   */
  HOST_DEV_INLINE bool IsSplittable(float minSplitLoss) const {
    return ((score >= minSplitLoss) && (index != INT_MAX));
  }
};

/**
 * @enum ArgMaxByKeyAlgo best_split_evaluation.cuh
 * @brief Help decide which algorithm to use for multi-argmax operation
 */
enum ArgMaxByKeyAlgo {
  /** simplest, use gmem-atomics for all updates */
  kAbkGmem = 0,
  /** use smem-atomics for updates (when number of keys are less) */
  kAbkSmem
};

/** max depth until which to use shared mem based atomics for argmax */
static const int kMaxAbkLevels = 3;

HOST_DEV_INLINE ExactSplitCandidate MaxSplit(ExactSplitCandidate a,
                                             ExactSplitCandidate b) {
  ExactSplitCandidate out;
  if (a.score < b.score) {
    out.score = b.score;
    out.index = b.index;
  } else if (a.score == b.score) {
    out.score = a.score;
    out.index = (a.index < b.index) ? a.index : b.index;
  } else {
    out.score = a.score;
    out.index = a.index;
  }
  return out;
}

DEV_INLINE void AtomicArgMax(ExactSplitCandidate* address,
                             ExactSplitCandidate val) {
  unsigned long long* intAddress = reinterpret_cast<unsigned long long*>(address);  // NOLINT
  unsigned long long old = *intAddress;                           // NOLINT
  unsigned long long assumed = old;                               // NOLINT
  do {
    assumed = old;
    ExactSplitCandidate res =
        MaxSplit(val, *reinterpret_cast<ExactSplitCandidate*>(&assumed));
    old = atomicCAS(intAddress, assumed, *reinterpret_cast<uint64_t*>(&res));
  } while (assumed != old);
}

DEV_INLINE void ArgMaxWithAtomics(
    int id,
    common::Span<ExactSplitCandidate> nodeSplits,
    common::Span<const GradientPair> gradScans,
    common::Span<const GradientPair> gradSums,
    common::Span<const float> vals,
    common::Span<const int> colIds,
    common::Span<const NodeIdT> nodeAssigns,
    common::Span<const DeviceNodeStats> nodes, int nUniqKeys,
    NodeIdT nodeStart, int len,
    const GPUTrainingParam& param) {
  int nodeId = nodeAssigns[id];
  // @todo: this is really a bad check! but will be fixed when we move
  //  to key-based reduction
  if ((id == 0) ||
      !((nodeId == nodeAssigns[id - 1]) && (colIds[id] == colIds[id - 1]) &&
        (vals[id] == vals[id - 1]))) {
    if (nodeId != kUnusedNode) {
      int sumId = Abs2UniqueKey(id, nodeAssigns, colIds, nodeStart, nUniqKeys);
      GradientPair colSum = gradSums[sumId];
      int uid = nodeId - nodeStart;
      DeviceNodeStats node_stat = nodes[nodeId];
      GradientPair parentSum = node_stat.sum_gradients;
      float parentGain = node_stat.root_gain;
      bool tmp;
      ExactSplitCandidate s;
      GradientPair missing = parentSum - colSum;
      s.score = LossChangeMissing(gradScans[id], missing, parentSum, parentGain,
                                  param, tmp);
      s.index = id;
      AtomicArgMax(&nodeSplits[uid], s);
    }  // end if nodeId != UNUSED_NODE
  }    // end if id == 0 ...
}

__global__ void AtomicArgMaxByKeyGmem(
    common::Span<ExactSplitCandidate> nodeSplits,
    common::Span<const GradientPair> gradScans,
    common::Span<const GradientPair> gradSums,
    common::Span<const float> vals,
    common::Span<const int> colIds,
    common::Span<const NodeIdT> nodeAssigns,
    common::Span<const DeviceNodeStats> nodes,
    int nUniqKeys,
    NodeIdT nodeStart,
    int len,
    const TrainParam param) {
  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  const int stride = blockDim.x * gridDim.x;
  for (; id < len; id += stride) {
    ArgMaxWithAtomics(id, nodeSplits, gradScans, gradSums, vals, colIds,
                      nodeAssigns, nodes, nUniqKeys, nodeStart, len,
                      GPUTrainingParam(param));
  }
}

__global__ void AtomicArgMaxByKeySmem(
    common::Span<ExactSplitCandidate> nodeSplits,
    common::Span<const GradientPair> gradScans,
    common::Span<const GradientPair> gradSums,
    common::Span<const float> vals,
    common::Span<const int> colIds,
    common::Span<const NodeIdT> nodeAssigns,
    common::Span<const DeviceNodeStats> nodes,
    int nUniqKeys, NodeIdT nodeStart, int len, const GPUTrainingParam param) {
  extern __shared__ char sArr[];
  common::Span<ExactSplitCandidate> sNodeSplits =
      common::Span<ExactSplitCandidate>(
          reinterpret_cast<ExactSplitCandidate*>(sArr),
          static_cast<typename common::Span<ExactSplitCandidate>::index_type>(
              nUniqKeys * sizeof(ExactSplitCandidate)));
  int tid = threadIdx.x;
  ExactSplitCandidate defVal;

  for (int i = tid; i < nUniqKeys; i += blockDim.x) {
    sNodeSplits[i] = defVal;
  }
  __syncthreads();
  int id = tid + (blockIdx.x * blockDim.x);
  const int stride = blockDim.x * gridDim.x;
  for (; id < len; id += stride) {
    ArgMaxWithAtomics(id, sNodeSplits, gradScans, gradSums, vals, colIds,
                      nodeAssigns, nodes, nUniqKeys, nodeStart, len, param);
  }
  __syncthreads();
  for (int i = tid; i < nUniqKeys; i += blockDim.x) {
    ExactSplitCandidate s = sNodeSplits[i];
    AtomicArgMax(&nodeSplits[i], s);
  }
}

/**
 * @brief Performs argmax_by_key functionality but for cases when keys need not
 *  occur contiguously
 * @param nodeSplits will contain information on best split for each node
 * @param gradScans exclusive sum on sorted segments for each col
 * @param gradSums gradient sum for each column in DMatrix based on to node-ids
 * @param vals feature values
 * @param colIds column index for each element in the feature values array
 * @param nodeAssigns node-id assignments to each element in DMatrix
 * @param nodes pointer to all nodes for this tree in BFS order
 * @param nUniqKeys number of unique node-ids in this level
 * @param nodeStart start index of the node-ids in this level
 * @param len number of elements
 * @param param training parameters
 * @param algo which algorithm to use for argmax_by_key
 */
template <int BLKDIM = 256, int ITEMS_PER_THREAD = 4>
void ArgMaxByKey(common::Span<ExactSplitCandidate> nodeSplits,
                 common::Span<const GradientPair> gradScans,
                 common::Span<const GradientPair> gradSums,
                 common::Span<const float> vals,
                 common::Span<const int> colIds,
                 common::Span<const NodeIdT> nodeAssigns,
                 common::Span<const DeviceNodeStats> nodes,
                 int nUniqKeys,
                 NodeIdT nodeStart, int len, const TrainParam param,
                 ArgMaxByKeyAlgo algo,
                 GPUSet const& devices) {
  dh::FillConst<ExactSplitCandidate, BLKDIM, ITEMS_PER_THREAD>(
      *(devices.begin()), nodeSplits.data(), nUniqKeys,
      ExactSplitCandidate());
  int nBlks = dh::DivRoundUp(len, ITEMS_PER_THREAD * BLKDIM);
  switch (algo) {
    case kAbkGmem:
      AtomicArgMaxByKeyGmem<<<nBlks, BLKDIM>>>(
          nodeSplits, gradScans, gradSums, vals, colIds, nodeAssigns, nodes,
          nUniqKeys, nodeStart, len, param);
      break;
    case kAbkSmem:
      AtomicArgMaxByKeySmem<<<nBlks, BLKDIM,
                              sizeof(ExactSplitCandidate) * nUniqKeys>>>(
          nodeSplits, gradScans, gradSums, vals, colIds, nodeAssigns, nodes,
          nUniqKeys, nodeStart, len, GPUTrainingParam(param));
      break;
    default:
      throw std::runtime_error("argMaxByKey: Bad algo passed!");
  }
}

__global__ void AssignColIds(int* colIds, const int* colOffsets) {
  int myId = blockIdx.x;
  int start = colOffsets[myId];
  int end = colOffsets[myId + 1];
  for (int id = start + threadIdx.x; id < end; id += blockDim.x) {
    colIds[id] = myId;
  }
}

__global__ void FillDefaultNodeIds(NodeIdT* nodeIdsPerInst,
                                   const DeviceNodeStats* nodes, int n_rows) {
  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  if (id >= n_rows) {
    return;
  }
  // if this element belongs to none of the currently active node-id's
  NodeIdT nId = nodeIdsPerInst[id];
  if (nId == kUnusedNode) {
    return;
  }
  const DeviceNodeStats n = nodes[nId];
  NodeIdT result;
  if (n.IsLeaf() || n.IsUnused()) {
    result = kUnusedNode;
  } else if (n.dir == kLeftDir) {
    result = (2 * n.idx) + 1;
  } else {
    result = (2 * n.idx) + 2;
  }
  nodeIdsPerInst[id] = result;
}

__global__ void AssignNodeIds(NodeIdT* nodeIdsPerInst, int* nodeLocations,
                              const NodeIdT* nodeIds, const int* instId,
                              const DeviceNodeStats* nodes,
                              const int* colOffsets, const float* vals,
                              int nVals, int nCols) {
  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  const int stride = blockDim.x * gridDim.x;
  for (; id < nVals; id += stride) {
    // fusing generation of indices for node locations
    nodeLocations[id] = id;
    // using nodeIds here since the previous kernel would have updated
    // the nodeIdsPerInst with all default assignments
    int nId = nodeIds[id];
    // if this element belongs to none of the currently active node-id's
    if (nId != kUnusedNode) {
      const DeviceNodeStats n = nodes[nId];
      int colId = n.fidx;
      // printf("nid=%d colId=%d id=%d\n", nId, colId, id);
      int start = colOffsets[colId];
      int end = colOffsets[colId + 1];
      // @todo: too much wasteful threads!!
      if ((id >= start) && (id < end) && !(n.IsLeaf() || n.IsUnused())) {
        NodeIdT result = (2 * n.idx) + 1 + (vals[id] >= n.fvalue);
        nodeIdsPerInst[instId[id]] = result;
      }
    }
  }
}

__global__ void MarkLeavesKernel(DeviceNodeStats* nodes, int len) {
  int id = (blockIdx.x * blockDim.x) + threadIdx.x;
  if ((id < len) && !nodes[id].IsUnused()) {
    int lid = (id << 1) + 1;
    int rid = (id << 1) + 2;
    if ((lid >= len) || (rid >= len)) {
      nodes[id].root_gain = -FLT_MAX;  // bottom-most nodes
    } else if (nodes[lid].IsUnused() && nodes[rid].IsUnused()) {
      nodes[id].root_gain = -FLT_MAX;  // unused child nodes
    }
  }
}

class GPUMaker : public TreeUpdater {
 protected:
  TrainParam param_;
  /** whether we have initialized memory already (so as not to repeat!) */
  bool allocated_;
  /** feature values stored in column-major compressed format */
  dh::DoubleBuffer<float> vals_;
  common::Span<float> vals_cached_;
  /** corresponding instance id's of these featutre values */
  dh::DoubleBuffer<int> instIds_;
  common::Span<int> inst_ids_cached_;
  /** column offsets for these feature values */
  common::Span<int> colOffsets_;
  common::Span<GradientPair> gradsInst_;
  dh::DoubleBuffer<NodeIdT> nodeAssigns_;
  dh::DoubleBuffer<int> nodeLocations_;
  common::Span<DeviceNodeStats> nodes_;
  common::Span<NodeIdT> node_assigns_per_inst_;
  common::Span<GradientPair> gradsums_;
  common::Span<GradientPair> gradscans_;
  common::Span<ExactSplitCandidate> nodeSplits_;
  int n_vals_;
  int n_rows_;
  int n_cols_;
  int maxNodes_;
  int maxLeaves_;

  // devices are only used for sharding the HostDeviceVector passed as a parameter;
  // the algorithm works with a single GPU only
  GPUSet devices_;

  dh::CubMemory tmp_mem_;
  common::Span<GradientPair> tmpScanGradBuff_;
  common::Span<int> tmp_scan_key_buff_;
  common::Span<int> colIds_;
  dh::BulkAllocator ba_;

 public:
  GPUMaker() : allocated_{false} {}
  ~GPUMaker() override = default;

  void Init(const std::vector<std::pair<std::string, std::string>> &args) override {
     param_.InitAllowUnknown(args);
     maxNodes_ = (1 << (param_.max_depth + 1)) - 1;
     maxLeaves_ = 1 << param_.max_depth;

     devices_ = GPUSet::All(tparam_->gpu_id, tparam_->n_gpus);
  }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) override {
    // rescale learning rate according to size of trees
    float lr = param_.learning_rate;
    param_.learning_rate = lr / trees.size();

    gpair->Shard(devices_);

    try {
      // build tree
      for (auto tree : trees) {
        UpdateTree(gpair, dmat, tree);
      }
    } catch (const std::exception& e) {
      LOG(FATAL) << "grow_gpu exception: " << e.what() << std::endl;
    }
    param_.learning_rate = lr;
  }
  /// @note: Update should be only after Init!!
  void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
                  RegTree* hTree) {
    if (!allocated_) {
      SetupOneTimeData(dmat);
    }
    for (int i = 0; i < param_.max_depth; ++i) {
      if (i == 0) {
        // make sure to start on a fresh tree with sorted values!
        dh::CopyDeviceSpan(vals_.CurrentSpan(), vals_cached_);
        dh::CopyDeviceSpan(instIds_.CurrentSpan(), inst_ids_cached_);
        TransferGrads(gpair);
      }
      int nNodes = 1 << i;
      NodeIdT nodeStart = nNodes - 1;
      InitNodeData(i, nodeStart, nNodes);
      FindSplit(i, nodeStart, nNodes);
    }
    // mark all the used nodes with unused children as leaf nodes
    MarkLeaves();
    Dense2SparseTree(hTree, nodes_, param_);
  }

  void Split2Node(int nNodes, NodeIdT nodeStart) {
    auto d_nodes = nodes_;
    auto d_gradScans = gradscans_;
    auto d_gradsums = gradsums_;
    auto d_nodeAssigns = nodeAssigns_.CurrentSpan();
    auto d_colIds = colIds_;
    auto d_vals = vals_.Current();
    auto d_nodeSplits = nodeSplits_.data();
    int nUniqKeys = nNodes;
    float min_split_loss = param_.min_split_loss;
    auto gpu_param = GPUTrainingParam(param_);

    dh::LaunchN(*(devices_.begin()), nNodes, [=] __device__(int uid) {
      int absNodeId = uid + nodeStart;
      ExactSplitCandidate s = d_nodeSplits[uid];
      if (s.IsSplittable(min_split_loss)) {
        int idx = s.index;
        int nodeInstId =
            Abs2UniqueKey(idx, d_nodeAssigns, d_colIds, nodeStart, nUniqKeys);
        bool missingLeft = true;
        const DeviceNodeStats& n = d_nodes[absNodeId];
        GradientPair gradScan = d_gradScans[idx];
        GradientPair gradSum = d_gradsums[nodeInstId];
        float thresh = d_vals[idx];
        int colId = d_colIds[idx];
        // get the default direction for the current node
        GradientPair missing = n.sum_gradients - gradSum;
        LossChangeMissing(gradScan, missing, n.sum_gradients, n.root_gain,
                          gpu_param, missingLeft);
        // get the score/weight/id/gradSum for left and right child nodes
        GradientPair lGradSum = missingLeft ? gradScan + missing : gradScan;
        GradientPair rGradSum = n.sum_gradients - lGradSum;

        // Create children
        d_nodes[LeftChildNodeIdx(absNodeId)] =
            DeviceNodeStats(lGradSum, LeftChildNodeIdx(absNodeId), gpu_param);
        d_nodes[RightChildNodeIdx(absNodeId)] =
            DeviceNodeStats(rGradSum, RightChildNodeIdx(absNodeId), gpu_param);
        // Set split for parent
        d_nodes[absNodeId].SetSplit(thresh, colId,
                                    missingLeft ? kLeftDir : kRightDir, lGradSum,
                                    rGradSum);
      } else {
        // cannot be split further, so this node is a leaf!
        d_nodes[absNodeId].root_gain = -FLT_MAX;
      }
    });
  }

  void FindSplit(int level, NodeIdT nodeStart, int nNodes) {
    ReduceScanByKey(gradsums_, gradscans_, gradsInst_,
                    instIds_.CurrentSpan(), nodeAssigns_.CurrentSpan(), n_vals_, nNodes,
                    n_cols_, tmpScanGradBuff_, tmp_scan_key_buff_,
                    colIds_, nodeStart);
    auto devices = GPUSet::All(tparam_->gpu_id, tparam_->n_gpus);
    ArgMaxByKey(nodeSplits_, gradscans_, gradsums_,
                vals_.CurrentSpan(), colIds_, nodeAssigns_.CurrentSpan(),
                nodes_, nNodes, nodeStart, n_vals_, param_,
                level <= kMaxAbkLevels ? kAbkSmem : kAbkGmem,
                devices);
    Split2Node(nNodes, nodeStart);
  }

  void AllocateAllData(int offsetSize) {
    int tmpBuffSize = ScanTempBufferSize(n_vals_);
    ba_.Allocate(*(devices_.begin()), &vals_, n_vals_,
                 &vals_cached_, n_vals_, &instIds_, n_vals_, &inst_ids_cached_, n_vals_,
                 &colOffsets_, offsetSize, &gradsInst_, n_rows_, &nodeAssigns_, n_vals_,
                 &nodeLocations_, n_vals_, &nodes_, maxNodes_, &node_assigns_per_inst_,
                 n_rows_, &gradsums_, maxLeaves_ * n_cols_, &gradscans_, n_vals_,
                 &nodeSplits_, maxLeaves_, &tmpScanGradBuff_, tmpBuffSize,
                 &tmp_scan_key_buff_, tmpBuffSize, &colIds_, n_vals_);
  }

  void SetupOneTimeData(DMatrix* dmat) {
    if (!dmat->SingleColBlock()) {
      LOG(FATAL) << "exact::GPUBuilder - must have 1 column block";
    }
    std::vector<float> fval;
    std::vector<int> fId;
    std::vector<int> offset;
    ConvertToCsc(dmat, &fval, &fId, &offset);
    AllocateAllData(static_cast<int>(offset.size()));
    TransferAndSortData(fval, fId, offset);
    allocated_ = true;
  }

  void ConvertToCsc(DMatrix* dmat, std::vector<float>* fval,
                    std::vector<int>* fId, std::vector<int>* offset) {
    const MetaInfo& info = dmat->Info();
    CHECK(info.num_col_ < std::numeric_limits<int>::max());
    CHECK(info.num_row_ < std::numeric_limits<int>::max());
    n_rows_ = static_cast<int>(info.num_row_);
    n_cols_ = static_cast<int>(info.num_col_);
    offset->reserve(n_cols_ + 1);
    offset->push_back(0);
    fval->reserve(n_cols_ * n_rows_);
    fId->reserve(n_cols_ * n_rows_);
    // in case you end up with a DMatrix having no column access
    // then make sure to enable that before copying the data!
    for (const auto& batch : dmat->GetSortedColumnBatches()) {
      for (int i = 0; i < batch.Size(); i++) {
        auto col = batch[i];
        for (const Entry& e : col) {
          int inst_id = static_cast<int>(e.index);
          fval->push_back(e.fvalue);
          fId->push_back(inst_id);
        }
        offset->push_back(static_cast<int>(fval->size()));
      }
    }
    CHECK(fval->size() < std::numeric_limits<int>::max());
    n_vals_ = static_cast<int>(fval->size());
  }

  void TransferAndSortData(const std::vector<float>& fval,
                           const std::vector<int>& fId,
                           const std::vector<int>& offset) {
    dh::CopyVectorToDeviceSpan(vals_.CurrentSpan(), fval);
    dh::CopyVectorToDeviceSpan(instIds_.CurrentSpan(), fId);
    dh::CopyVectorToDeviceSpan(colOffsets_, offset);
    dh::SegmentedSort<float, int>(&tmp_mem_, &vals_, &instIds_, n_vals_, n_cols_,
                                  colOffsets_);
    dh::CopyDeviceSpan(vals_cached_, vals_.CurrentSpan());
    dh::CopyDeviceSpan(inst_ids_cached_, instIds_.CurrentSpan());
    AssignColIds<<<n_cols_, 512>>>(colIds_.data(), colOffsets_.data());
  }

  void TransferGrads(HostDeviceVector<GradientPair>* gpair) {
    gpair->GatherTo(
        thrust::device_pointer_cast(gradsInst_.data()),
        thrust::device_pointer_cast(gradsInst_.data() + gradsInst_.size()));
    // evaluate the full-grad reduction for the root node
    dh::SumReduction<GradientPair>(tmp_mem_, gradsInst_, gradsums_, n_rows_);
  }

  void InitNodeData(int level, NodeIdT nodeStart, int nNodes) {
    // all instances belong to root node at the beginning!
    if (level == 0) {
      thrust::fill(thrust::device_pointer_cast(nodes_.data()),
                   thrust::device_pointer_cast(nodes_.data() + nodes_.size()),
                   DeviceNodeStats());
      thrust::fill(thrust::device_pointer_cast(nodeAssigns_.Current()),
                   thrust::device_pointer_cast(nodeAssigns_.Current() +
                                               nodeAssigns_.Size()),
                   0);
      thrust::fill(thrust::device_pointer_cast(node_assigns_per_inst_.data()),
                   thrust::device_pointer_cast(node_assigns_per_inst_.data() +
                                               node_assigns_per_inst_.size()),
                   0);
      // for root node, just update the gradient/score/weight/id info
      // before splitting it! Currently all data is on GPU, hence this
      // stupid little kernel
      auto d_nodes = nodes_;
      auto d_sums = gradsums_;
      auto gpu_params = GPUTrainingParam(param_);
      dh::LaunchN(*(devices_.begin()), 1, [=] __device__(int idx) {
        d_nodes[0] = DeviceNodeStats(d_sums[0], 0, gpu_params);
      });
    } else {
      const int BlkDim = 256;
      const int ItemsPerThread = 4;
      // assign default node ids first
      int nBlks = dh::DivRoundUp(n_rows_, BlkDim);
      FillDefaultNodeIds<<<nBlks, BlkDim>>>(node_assigns_per_inst_.data(),
                                            nodes_.data(), n_rows_);
      // evaluate the correct child indices of non-missing values next
      nBlks = dh::DivRoundUp(n_vals_, BlkDim * ItemsPerThread);
      AssignNodeIds<<<nBlks, BlkDim>>>(
          node_assigns_per_inst_.data(), nodeLocations_.Current(),
          nodeAssigns_.Current(), instIds_.Current(), nodes_.data(),
          colOffsets_.data(), vals_.Current(), n_vals_, n_cols_);
      // gather the node assignments across all other columns too
      dh::Gather(*(devices_.begin()), nodeAssigns_.Current(),
                 node_assigns_per_inst_.data(), instIds_.Current(), n_vals_);
      SortKeys(level);
    }
  }

  void SortKeys(int level) {
    // segmented-sort the arrays based on node-id's
    // but we don't need more than level+1 bits for sorting!
    SegmentedSort(&tmp_mem_, &nodeAssigns_, &nodeLocations_, n_vals_, n_cols_,
                  colOffsets_, 0, level + 1);
    dh::Gather<float, int>(*(devices_.begin()), vals_.other(),
                           vals_.Current(), instIds_.other(), instIds_.Current(),
                           nodeLocations_.Current(), n_vals_);
    vals_.buff.selector ^= 1;
    instIds_.buff.selector ^= 1;
  }

  void MarkLeaves() {
    const int BlkDim = 128;
    int nBlks = dh::DivRoundUp(maxNodes_, BlkDim);
    MarkLeavesKernel<<<nBlks, BlkDim>>>(nodes_.data(), maxNodes_);
  }
};

XGBOOST_REGISTER_TREE_UPDATER(GPUMaker, "grow_gpu")
    .describe("Grow tree with GPU.")
    .set_body([]() { return new GPUMaker(); });

}  // namespace tree
}  // namespace xgboost
