#include "hip/hip_runtime.h"
/*!
 * Copyright 2017 XGBoost contributors
 */
#include <xgboost/tree_updater.h>
#include <utility>
#include <vector>
#include "param.h"
#include "updater_gpu_common.cuh"

namespace xgboost {
namespace tree {

DMLC_REGISTRY_FILE_TAG(updater_gpu);

/**
 * @brief Absolute BFS order IDs to col-wise unique IDs based on user input
 * @param tid the index of the element that this thread should access
 * @param abs the array of absolute IDs
 * @param colIds the array of column IDs for each element
 * @param nodeStart the start of the node ID at this level
 * @param nKeys number of nodes at this level.
 * @return the uniq key
 */

static HOST_DEV_INLINE NodeIdT abs2uniqKey(int tid, const NodeIdT* abs,
                                             const int* colIds,
                                             NodeIdT nodeStart, int nKeys) {
  int a = abs[tid];
  if (a == kUnusedNode) return a;
  return ((a - nodeStart) + (colIds[tid] * nKeys));
}

/**
 * @struct Pair
 * @brief Pair used for key basd scan operations on GradientPair
 */
struct Pair {
  int key;
  GradientPair value;
};

/** define a key that's not used at all in the entire boosting process */
static const int kNoneKey = -100;

/**
 * @brief Allocate temporary buffers needed for scan operations
 * @param tmpScans gradient buffer
 * @param tmpKeys keys buffer
 * @param size number of elements that will be scanned
 */
template <int BLKDIM_L1L3 = 256>
int ScanTempBufferSize(int size) {
  int num_blocks = dh::DivRoundUp(size, BLKDIM_L1L3);
  return num_blocks;
}

struct AddByKey {
  template <typename T>
  HOST_DEV_INLINE T operator()(const T& first, const T& second) const {
    T result;
    if (first.key == second.key) {
      result.key = first.key;
      result.value = first.value + second.value;
    } else {
      result.key = second.key;
      result.value = second.value;
    }
    return result;
  }
};

/**
 * @brief Gradient value getter function
 * @param id the index into the vals or instIds array to which to fetch
 * @param vals the gradient value buffer
 * @param instIds instance index buffer
 * @return the expected gradient value
 */
HOST_DEV_INLINE GradientPair get(int id, const GradientPair* vals,
                              const int* instIds) {
  id = instIds[id];
  return vals[id];
}

template <int BLKDIM_L1L3>
__global__ void cubScanByKeyL1(GradientPair* scans, const GradientPair* vals,
                               const int* instIds, GradientPair* mScans,
                               int* mKeys, const NodeIdT* keys, int nUniqKeys,
                               const int* colIds, NodeIdT nodeStart,
                               const int size) {
  Pair rootPair = {kNoneKey, GradientPair(0.f, 0.f)};
  int myKey;
  GradientPair myValue;
  typedef hipcub::BlockScan<Pair, BLKDIM_L1L3> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  Pair threadData;
  int tid = blockIdx.x * BLKDIM_L1L3 + threadIdx.x;
  if (tid < size) {
    myKey = abs2uniqKey(tid, keys, colIds, nodeStart, nUniqKeys);
    myValue = get(tid, vals, instIds);
  } else {
    myKey = kNoneKey;
    myValue = {};
  }
  threadData.key = myKey;
  threadData.value = myValue;
  // get previous key, especially needed for the last thread in this block
  // in order to pass on the partial scan values.
  // this statement MUST appear before the checks below!
  // else, the result of this shuffle operation will be undefined
  int previousKey = __shfl_up(myKey, 1);
  // Collectively compute the block-wide exclusive prefix sum
  BlockScan(temp_storage)
      .ExclusiveScan(threadData, threadData, rootPair, AddByKey());
  if (tid < size) {
    scans[tid] = threadData.value;
  } else {
    return;
  }
  if (threadIdx.x == BLKDIM_L1L3 - 1) {
    threadData.value =
        (myKey == previousKey) ? threadData.value : GradientPair(0.0f, 0.0f);
    mKeys[blockIdx.x] = myKey;
    mScans[blockIdx.x] = threadData.value + myValue;
  }
}

template <int BLKSIZE>
__global__ void cubScanByKeyL2(GradientPair* mScans, int* mKeys, int mLength) {
  typedef hipcub::BlockScan<Pair, BLKSIZE, hipcub::BLOCK_SCAN_WARP_SCANS> BlockScan;
  Pair threadData;
  __shared__ typename BlockScan::TempStorage temp_storage;
  for (int i = threadIdx.x; i < mLength; i += BLKSIZE - 1) {
    threadData.key = mKeys[i];
    threadData.value = mScans[i];
    BlockScan(temp_storage).InclusiveScan(threadData, threadData, AddByKey());
    mScans[i] = threadData.value;
    __syncthreads();
  }
}

template <int BLKDIM_L1L3>
__global__ void cubScanByKeyL3(GradientPair* sums, GradientPair* scans,
                               const GradientPair* vals, const int* instIds,
                               const GradientPair* mScans, const int* mKeys,
                               const NodeIdT* keys, int nUniqKeys,
                               const int* colIds, NodeIdT nodeStart,
                               const int size) {
  int relId = threadIdx.x;
  int tid = (blockIdx.x * BLKDIM_L1L3) + relId;
  // to avoid the following warning from nvcc:
  //   __shared__ memory variable with non-empty constructor or destructor
  //     (potential race between threads)
  __shared__ char gradBuff[sizeof(GradientPair)];
  __shared__ int s_mKeys;
  GradientPair* s_mScans = reinterpret_cast<GradientPair*>(gradBuff);
  if (tid >= size) return;
  // cache block-wide partial scan info
  if (relId == 0) {
    s_mKeys = (blockIdx.x > 0) ? mKeys[blockIdx.x - 1] : kNoneKey;
    s_mScans[0] = (blockIdx.x > 0) ? mScans[blockIdx.x - 1] : GradientPair();
  }
  int myKey = abs2uniqKey(tid, keys, colIds, nodeStart, nUniqKeys);
  int previousKey =
      tid == 0 ? kNoneKey
               : abs2uniqKey(tid - 1, keys, colIds, nodeStart, nUniqKeys);
  GradientPair myValue = scans[tid];
  __syncthreads();
  if (blockIdx.x > 0 && s_mKeys == previousKey) {
    myValue += s_mScans[0];
  }
  if (tid == size - 1) {
    sums[previousKey] = myValue + get(tid, vals, instIds);
  }
  if ((previousKey != myKey) && (previousKey >= 0)) {
    sums[previousKey] = myValue;
    myValue = GradientPair(0.0f, 0.0f);
  }
  scans[tid] = myValue;
}

/**
 * @brief Performs fused reduce and scan by key functionality. It is assumed
 * that
 *  the keys occur contiguously!
 * @param sums the output gradient reductions for each element performed
 * key-wise
 * @param scans the output gradient scans for each element performed key-wise
 * @param vals the gradients evaluated for each observation.
 * @param instIds instance ids for each element
 * @param keys keys to be used to segment the reductions. They need not occur
 *  contiguously in contrast to scan_by_key. Currently, we need one key per
 *  value in the 'vals' array.
 * @param size number of elements in the 'vals' array
 * @param nUniqKeys max number of uniq keys found per column
 * @param nCols number of columns
 * @param tmpScans temporary scan buffer needed for cub-pyramid algo
 * @param tmpKeys temporary key buffer needed for cub-pyramid algo
 * @param colIds column indices for each element in the array
 * @param nodeStart index of the leftmost node in the current level
 */
template <int BLKDIM_L1L3 = 256, int BLKDIM_L2 = 512>
void reduceScanByKey(GradientPair* sums, GradientPair* scans, const GradientPair* vals,
                     const int* instIds, const NodeIdT* keys, int size,
                     int nUniqKeys, int nCols, GradientPair* tmpScans,
                     int* tmpKeys, const int* colIds, NodeIdT nodeStart) {
  int nBlks = dh::DivRoundUp(size, BLKDIM_L1L3);
  hipMemset(sums, 0, nUniqKeys * nCols * sizeof(GradientPair));
  cubScanByKeyL1<BLKDIM_L1L3>
      <<<nBlks, BLKDIM_L1L3>>>(scans, vals, instIds, tmpScans, tmpKeys, keys,
                               nUniqKeys, colIds, nodeStart, size);
  cubScanByKeyL2<BLKDIM_L2><<<1, BLKDIM_L2>>>(tmpScans, tmpKeys, nBlks);
  cubScanByKeyL3<BLKDIM_L1L3>
      <<<nBlks, BLKDIM_L1L3>>>(sums, scans, vals, instIds, tmpScans, tmpKeys,
                               keys, nUniqKeys, colIds, nodeStart, size);
}

/**
 * @struct ExactSplitCandidate
 * @brief Abstraction of a possible split in the decision tree
 */
struct ExactSplitCandidate {
  /** the optimal gain score for this node */
  float score;
  /** index where to split in the DMatrix */
  int index;

  HOST_DEV_INLINE ExactSplitCandidate() : score(-FLT_MAX), index(INT_MAX) {}

  /**
   * @brief Whether the split info is valid to be used to create a new child
   * @param minSplitLoss minimum score above which decision to split is made
   * @return true if splittable, else false
   */
  HOST_DEV_INLINE bool isSplittable(float minSplitLoss) const {
    return ((score >= minSplitLoss) && (index != INT_MAX));
  }
};

/**
 * @enum ArgMaxByKeyAlgo best_split_evaluation.cuh
 * @brief Help decide which algorithm to use for multi-argmax operation
 */
enum ArgMaxByKeyAlgo {
  /** simplest, use gmem-atomics for all updates */
  kAbkGmem = 0,
  /** use smem-atomics for updates (when number of keys are less) */
  kAbkSmem
};

/** max depth until which to use shared mem based atomics for argmax */
static const int kMaxAbkLevels = 3;

HOST_DEV_INLINE ExactSplitCandidate maxSplit(ExactSplitCandidate a,
                                             ExactSplitCandidate b) {
  ExactSplitCandidate out;
  if (a.score < b.score) {
    out.score = b.score;
    out.index = b.index;
  } else if (a.score == b.score) {
    out.score = a.score;
    out.index = (a.index < b.index) ? a.index : b.index;
  } else {
    out.score = a.score;
    out.index = a.index;
  }
  return out;
}

DEV_INLINE void atomicArgMax(ExactSplitCandidate* address,
                             ExactSplitCandidate val) {
  unsigned long long* intAddress = (unsigned long long*)address;  // NOLINT
  unsigned long long old = *intAddress;                           // NOLINT
  unsigned long long assumed;                                     // NOLINT
  do {
    assumed = old;
    ExactSplitCandidate res =
        maxSplit(val, *reinterpret_cast<ExactSplitCandidate*>(&assumed));
    old = atomicCAS(intAddress, assumed, *reinterpret_cast<uint64_t*>(&res));
  } while (assumed != old);
}

DEV_INLINE void argMaxWithAtomics(
    int id, ExactSplitCandidate* nodeSplits, const GradientPair* gradScans,
    const GradientPair* gradSums, const float* vals, const int* colIds,
    const NodeIdT* nodeAssigns, const DeviceNodeStats* nodes, int nUniqKeys,
    NodeIdT nodeStart, int len, const GPUTrainingParam& param) {
  int nodeId = nodeAssigns[id];
  // @todo: this is really a bad check! but will be fixed when we move
  //  to key-based reduction
  if ((id == 0) ||
      !((nodeId == nodeAssigns[id - 1]) && (colIds[id] == colIds[id - 1]) &&
        (vals[id] == vals[id - 1]))) {
    if (nodeId != kUnusedNode) {
      int sumId = abs2uniqKey(id, nodeAssigns, colIds, nodeStart, nUniqKeys);
      GradientPair colSum = gradSums[sumId];
      int uid = nodeId - nodeStart;
      DeviceNodeStats n = nodes[nodeId];
      GradientPair parentSum = n.sum_gradients;
      float parentGain = n.root_gain;
      bool tmp;
      ExactSplitCandidate s;
      GradientPair missing = parentSum - colSum;
      s.score = LossChangeMissing(gradScans[id], missing, parentSum, parentGain,
                                 param, tmp);
      s.index = id;
      atomicArgMax(nodeSplits + uid, s);
    }  // end if nodeId != UNUSED_NODE
  }    // end if id == 0 ...
}

__global__ void atomicArgMaxByKeyGmem(
    ExactSplitCandidate* nodeSplits, const GradientPair* gradScans,
    const GradientPair* gradSums, const float* vals, const int* colIds,
    const NodeIdT* nodeAssigns, const DeviceNodeStats* nodes, int nUniqKeys,
    NodeIdT nodeStart, int len, const TrainParam param) {
  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  const int stride = blockDim.x * gridDim.x;
  for (; id < len; id += stride) {
    argMaxWithAtomics(id, nodeSplits, gradScans, gradSums, vals, colIds,
                      nodeAssigns, nodes, nUniqKeys, nodeStart, len,
                      GPUTrainingParam(param));
  }
}

__global__ void atomicArgMaxByKeySmem(
    ExactSplitCandidate* nodeSplits, const GradientPair* gradScans,
    const GradientPair* gradSums, const float* vals, const int* colIds,
    const NodeIdT* nodeAssigns, const DeviceNodeStats* nodes, int nUniqKeys,
    NodeIdT nodeStart, int len, const GPUTrainingParam param) {
  extern __shared__ char sArr[];
  ExactSplitCandidate* sNodeSplits =
      reinterpret_cast<ExactSplitCandidate*>(sArr);
  int tid = threadIdx.x;
  ExactSplitCandidate defVal;
#pragma unroll 1
  for (int i = tid; i < nUniqKeys; i += blockDim.x) {
    sNodeSplits[i] = defVal;
  }
  __syncthreads();
  int id = tid + (blockIdx.x * blockDim.x);
  const int stride = blockDim.x * gridDim.x;
  for (; id < len; id += stride) {
    argMaxWithAtomics(id, sNodeSplits, gradScans, gradSums, vals, colIds,
                      nodeAssigns, nodes, nUniqKeys, nodeStart, len, param);
  }
  __syncthreads();
  for (int i = tid; i < nUniqKeys; i += blockDim.x) {
    ExactSplitCandidate s = sNodeSplits[i];
    atomicArgMax(nodeSplits + i, s);
  }
}

/**
 * @brief Performs argmax_by_key functionality but for cases when keys need not
 *  occur contiguously
 * @param nodeSplits will contain information on best split for each node
 * @param gradScans exclusive sum on sorted segments for each col
 * @param gradSums gradient sum for each column in DMatrix based on to node-ids
 * @param vals feature values
 * @param colIds column index for each element in the feature values array
 * @param nodeAssigns node-id assignments to each element in DMatrix
 * @param nodes pointer to all nodes for this tree in BFS order
 * @param nUniqKeys number of unique node-ids in this level
 * @param nodeStart start index of the node-ids in this level
 * @param len number of elements
 * @param param training parameters
 * @param algo which algorithm to use for argmax_by_key
 */
template <int BLKDIM = 256, int ITEMS_PER_THREAD = 4>
void argMaxByKey(ExactSplitCandidate* nodeSplits, const GradientPair* gradScans,
                 const GradientPair* gradSums, const float* vals,
                 const int* colIds, const NodeIdT* nodeAssigns,
                 const DeviceNodeStats* nodes, int nUniqKeys,
                 NodeIdT nodeStart, int len, const TrainParam param,
                 ArgMaxByKeyAlgo algo) {
  dh::FillConst<ExactSplitCandidate, BLKDIM, ITEMS_PER_THREAD>(
      dh::GetDeviceIdx(param.gpu_id), nodeSplits, nUniqKeys,
      ExactSplitCandidate());
  int nBlks = dh::DivRoundUp(len, ITEMS_PER_THREAD * BLKDIM);
  switch (algo) {
    case kAbkGmem:
      atomicArgMaxByKeyGmem<<<nBlks, BLKDIM>>>(
          nodeSplits, gradScans, gradSums, vals, colIds, nodeAssigns, nodes,
          nUniqKeys, nodeStart, len, param);
      break;
    case kAbkSmem:
      atomicArgMaxByKeySmem<<<nBlks, BLKDIM,
                              sizeof(ExactSplitCandidate) * nUniqKeys>>>(
          nodeSplits, gradScans, gradSums, vals, colIds, nodeAssigns, nodes,
          nUniqKeys, nodeStart, len, GPUTrainingParam(param));
      break;
    default:
      throw std::runtime_error("argMaxByKey: Bad algo passed!");
  }
}

__global__ void assignColIds(int* colIds, const int* colOffsets) {
  int myId = blockIdx.x;
  int start = colOffsets[myId];
  int end = colOffsets[myId + 1];
  for (int id = start + threadIdx.x; id < end; id += blockDim.x) {
    colIds[id] = myId;
  }
}

__global__ void fillDefaultNodeIds(NodeIdT* nodeIdsPerInst,
                                   const DeviceNodeStats* nodes, int nRows) {
  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  if (id >= nRows) {
    return;
  }
  // if this element belongs to none of the currently active node-id's
  NodeIdT nId = nodeIdsPerInst[id];
  if (nId == kUnusedNode) {
    return;
  }
  const DeviceNodeStats n = nodes[nId];
  NodeIdT result;
  if (n.IsLeaf() || n.IsUnused()) {
    result = kUnusedNode;
  } else if (n.dir == kLeftDir) {
    result = (2 * n.idx) + 1;
  } else {
    result = (2 * n.idx) + 2;
  }
  nodeIdsPerInst[id] = result;
}

__global__ void assignNodeIds(NodeIdT* nodeIdsPerInst, int* nodeLocations,
                              const NodeIdT* nodeIds, const int* instId,
                              const DeviceNodeStats* nodes,
                              const int* colOffsets, const float* vals,
                              int nVals, int nCols) {
  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  const int stride = blockDim.x * gridDim.x;
  for (; id < nVals; id += stride) {
    // fusing generation of indices for node locations
    nodeLocations[id] = id;
    // using nodeIds here since the previous kernel would have updated
    // the nodeIdsPerInst with all default assignments
    int nId = nodeIds[id];
    // if this element belongs to none of the currently active node-id's
    if (nId != kUnusedNode) {
      const DeviceNodeStats n = nodes[nId];
      int colId = n.fidx;
      // printf("nid=%d colId=%d id=%d\n", nId, colId, id);
      int start = colOffsets[colId];
      int end = colOffsets[colId + 1];
      // @todo: too much wasteful threads!!
      if ((id >= start) && (id < end) && !(n.IsLeaf() || n.IsUnused())) {
        NodeIdT result = (2 * n.idx) + 1 + (vals[id] >= n.fvalue);
        nodeIdsPerInst[instId[id]] = result;
      }
    }
  }
}

__global__ void markLeavesKernel(DeviceNodeStats* nodes, int len) {
  int id = (blockIdx.x * blockDim.x) + threadIdx.x;
  if ((id < len) && !nodes[id].IsUnused()) {
    int lid = (id << 1) + 1;
    int rid = (id << 1) + 2;
    if ((lid >= len) || (rid >= len)) {
      nodes[id].root_gain = -FLT_MAX;  // bottom-most nodes
    } else if (nodes[lid].IsUnused() && nodes[rid].IsUnused()) {
      nodes[id].root_gain = -FLT_MAX;  // unused child nodes
    }
  }
}

class GPUMaker : public TreeUpdater {
 protected:
  TrainParam param;
  /** whether we have initialized memory already (so as not to repeat!) */
  bool allocated;
  /** feature values stored in column-major compressed format */
  dh::DVec2<float> vals;
  dh::DVec<float> vals_cached;
  /** corresponding instance id's of these featutre values */
  dh::DVec2<int> instIds;
  dh::DVec<int> instIds_cached;
  /** column offsets for these feature values */
  dh::DVec<int> colOffsets;
  dh::DVec<GradientPair> gradsInst;
  dh::DVec2<NodeIdT> nodeAssigns;
  dh::DVec2<int> nodeLocations;
  dh::DVec<DeviceNodeStats> nodes;
  dh::DVec<NodeIdT> nodeAssignsPerInst;
  dh::DVec<GradientPair> gradSums;
  dh::DVec<GradientPair> gradScans;
  dh::DVec<ExactSplitCandidate> nodeSplits;
  int nVals;
  int nRows;
  int nCols;
  int maxNodes;
  int maxLeaves;

  // devices are only used for resharding the HostDeviceVector passed as a parameter;
  // the algorithm works with a single GPU only
  GPUSet devices;

  dh::CubMemory tmp_mem;
  dh::DVec<GradientPair> tmpScanGradBuff;
  dh::DVec<int> tmpScanKeyBuff;
  dh::DVec<int> colIds;
  dh::BulkAllocator<dh::MemoryType::kDevice> ba;

 public:
  GPUMaker() : allocated(false) {}
  ~GPUMaker() {}

  void Init(
      const std::vector<std::pair<std::string, std::string>>& args) override {
    param.InitAllowUnknown(args);
    maxNodes = (1 << (param.max_depth + 1)) - 1;
    maxLeaves = 1 << param.max_depth;

    devices = GPUSet::Range(param.gpu_id, dh::NDevicesAll(param.n_gpus));
  }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) override {
    GradStats::CheckInfo(dmat->Info());
    // rescale learning rate according to size of trees
    float lr = param.learning_rate;
    param.learning_rate = lr / trees.size();

    gpair->Reshard(devices);

    try {
      // build tree
      for (size_t i = 0; i < trees.size(); ++i) {
        UpdateTree(gpair, dmat, trees[i]);
      }
    } catch (const std::exception& e) {
      LOG(FATAL) << "GPU plugin exception: " << e.what() << std::endl;
    }
    param.learning_rate = lr;
  }
  /// @note: Update should be only after Init!!
  void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
                  RegTree* hTree) {
    if (!allocated) {
      setupOneTimeData(dmat);
    }
    for (int i = 0; i < param.max_depth; ++i) {
      if (i == 0) {
        // make sure to start on a fresh tree with sorted values!
        vals.CurrentDVec() = vals_cached;
        instIds.CurrentDVec() = instIds_cached;
        transferGrads(gpair);
      }
      int nNodes = 1 << i;
      NodeIdT nodeStart = nNodes - 1;
      initNodeData(i, nodeStart, nNodes);
      findSplit(i, nodeStart, nNodes);
    }
    // mark all the used nodes with unused children as leaf nodes
    markLeaves();
    Dense2SparseTree(hTree, nodes, param);
  }

  void split2node(int nNodes, NodeIdT nodeStart) {
    auto d_nodes = nodes.Data();
    auto d_gradScans = gradScans.Data();
    auto d_gradSums = gradSums.Data();
    auto d_nodeAssigns = nodeAssigns.Current();
    auto d_colIds = colIds.Data();
    auto d_vals = vals.Current();
    auto d_nodeSplits = nodeSplits.Data();
    int nUniqKeys = nNodes;
    float min_split_loss = param.min_split_loss;
    auto gpu_param = GPUTrainingParam(param);

    dh::LaunchN(param.gpu_id, nNodes, [=] __device__(int uid) {
      int absNodeId = uid + nodeStart;
      ExactSplitCandidate s = d_nodeSplits[uid];
      if (s.isSplittable(min_split_loss)) {
        int idx = s.index;
        int nodeInstId =
            abs2uniqKey(idx, d_nodeAssigns, d_colIds, nodeStart, nUniqKeys);
        bool missingLeft = true;
        const DeviceNodeStats& n = d_nodes[absNodeId];
        GradientPair gradScan = d_gradScans[idx];
        GradientPair gradSum = d_gradSums[nodeInstId];
        float thresh = d_vals[idx];
        int colId = d_colIds[idx];
        // get the default direction for the current node
        GradientPair missing = n.sum_gradients - gradSum;
        LossChangeMissing(gradScan, missing, n.sum_gradients, n.root_gain,
                         gpu_param, missingLeft);
        // get the score/weight/id/gradSum for left and right child nodes
        GradientPair lGradSum = missingLeft ? gradScan + missing : gradScan;
        GradientPair rGradSum = n.sum_gradients - lGradSum;

        // Create children
        d_nodes[LeftChildNodeIdx(absNodeId)] =
            DeviceNodeStats(lGradSum, LeftChildNodeIdx(absNodeId), gpu_param);
        d_nodes[RightChildNodeIdx(absNodeId)] =
            DeviceNodeStats(rGradSum, RightChildNodeIdx(absNodeId), gpu_param);
        // Set split for parent
        d_nodes[absNodeId].SetSplit(thresh, colId,
                                    missingLeft ? kLeftDir : kRightDir, lGradSum,
                                    rGradSum);
      } else {
        // cannot be split further, so this node is a leaf!
        d_nodes[absNodeId].root_gain = -FLT_MAX;
      }
    });
  }

  void findSplit(int level, NodeIdT nodeStart, int nNodes) {
    reduceScanByKey(gradSums.Data(), gradScans.Data(), gradsInst.Data(),
                    instIds.Current(), nodeAssigns.Current(), nVals, nNodes,
                    nCols, tmpScanGradBuff.Data(), tmpScanKeyBuff.Data(),
                    colIds.Data(), nodeStart);
    argMaxByKey(nodeSplits.Data(), gradScans.Data(), gradSums.Data(),
                vals.Current(), colIds.Data(), nodeAssigns.Current(),
                nodes.Data(), nNodes, nodeStart, nVals, param,
                level <= kMaxAbkLevels ? kAbkSmem : kAbkGmem);
    split2node(nNodes, nodeStart);
  }

  void allocateAllData(int offsetSize) {
    int tmpBuffSize = ScanTempBufferSize(nVals);
    ba.Allocate(dh::GetDeviceIdx(param.gpu_id), param.silent, &vals, nVals,
                &vals_cached, nVals, &instIds, nVals, &instIds_cached, nVals,
                &colOffsets, offsetSize, &gradsInst, nRows, &nodeAssigns, nVals,
                &nodeLocations, nVals, &nodes, maxNodes, &nodeAssignsPerInst,
                nRows, &gradSums, maxLeaves * nCols, &gradScans, nVals,
                &nodeSplits, maxLeaves, &tmpScanGradBuff, tmpBuffSize,
                &tmpScanKeyBuff, tmpBuffSize, &colIds, nVals);
  }

  void setupOneTimeData(DMatrix* dmat) {
    size_t free_memory = dh::AvailableMemory(dh::GetDeviceIdx(param.gpu_id));
    if (!dmat->SingleColBlock()) {
      throw std::runtime_error("exact::GPUBuilder - must have 1 column block");
    }
    std::vector<float> fval;
    std::vector<int> fId;
    std::vector<size_t> offset;
    convertToCsc(dmat, &fval, &fId, &offset);
    allocateAllData(static_cast<int>(offset.size()));
    transferAndSortData(fval, fId, offset);
    allocated = true;
  }

  void convertToCsc(DMatrix* dmat, std::vector<float>* fval,
                    std::vector<int>* fId, std::vector<size_t>* offset) {
    MetaInfo info = dmat->Info();
    CHECK(info.num_col_ < std::numeric_limits<int>::max());
    CHECK(info.num_row_ < std::numeric_limits<int>::max());
    nRows = static_cast<int>(info.num_row_);
    nCols = static_cast<int>(info.num_col_);
    offset->reserve(nCols + 1);
    offset->push_back(0);
    fval->reserve(nCols * nRows);
    fId->reserve(nCols * nRows);
    // in case you end up with a DMatrix having no column access
    // then make sure to enable that before copying the data!
    if (!dmat->HaveColAccess(true)) {
      dmat->InitColAccess(nRows, true);
    }
    auto iter = dmat->ColIterator();
    iter->BeforeFirst();
    while (iter->Next()) {
      auto &batch = iter->Value();
      for (int i = 0; i < batch.Size(); i++) {
        auto col = batch[i];
        for (const Entry* it = col.data; it != col.data + col.length;
             it++) {
          int inst_id = static_cast<int>(it->index);
          fval->push_back(it->fvalue);
          fId->push_back(inst_id);
        }
        offset->push_back(fval->size());
      }
    }
    CHECK(fval->size() < std::numeric_limits<int>::max());
    nVals = static_cast<int>(fval->size());
  }

  void transferAndSortData(const std::vector<float>& fval,
                           const std::vector<int>& fId,
                           const std::vector<size_t>& offset) {
    vals.CurrentDVec() = fval;
    instIds.CurrentDVec() = fId;
    colOffsets = offset;
    dh::SegmentedSort<float, int>(&tmp_mem, &vals, &instIds, nVals, nCols,
                                  colOffsets);
    vals_cached = vals.CurrentDVec();
    instIds_cached = instIds.CurrentDVec();
    assignColIds<<<nCols, 512>>>(colIds.Data(), colOffsets.Data());
  }

  void transferGrads(HostDeviceVector<GradientPair>* gpair) {
    gpair->GatherTo(gradsInst.tbegin(), gradsInst.tend());
    // evaluate the full-grad reduction for the root node
    dh::SumReduction<GradientPair>(tmp_mem, gradsInst, gradSums, nRows);
  }

  void initNodeData(int level, NodeIdT nodeStart, int nNodes) {
    // all instances belong to root node at the beginning!
    if (level == 0) {
      nodes.Fill(DeviceNodeStats());
      nodeAssigns.CurrentDVec().Fill(0);
      nodeAssignsPerInst.Fill(0);
      // for root node, just update the gradient/score/weight/id info
      // before splitting it! Currently all data is on GPU, hence this
      // stupid little kernel
      auto d_nodes = nodes.Data();
      auto d_sums = gradSums.Data();
      auto gpu_params = GPUTrainingParam(param);
      dh::LaunchN(param.gpu_id, 1, [=] __device__(int idx) {
        d_nodes[0] = DeviceNodeStats(d_sums[0], 0, gpu_params);
      });
    } else {
      const int BlkDim = 256;
      const int ItemsPerThread = 4;
      // assign default node ids first
      int nBlks = dh::DivRoundUp(nRows, BlkDim);
      fillDefaultNodeIds<<<nBlks, BlkDim>>>(nodeAssignsPerInst.Data(),
                                            nodes.Data(), nRows);
      // evaluate the correct child indices of non-missing values next
      nBlks = dh::DivRoundUp(nVals, BlkDim * ItemsPerThread);
      assignNodeIds<<<nBlks, BlkDim>>>(
          nodeAssignsPerInst.Data(), nodeLocations.Current(),
          nodeAssigns.Current(), instIds.Current(), nodes.Data(),
          colOffsets.Data(), vals.Current(), nVals, nCols);
      // gather the node assignments across all other columns too
      dh::Gather(dh::GetDeviceIdx(param.gpu_id), nodeAssigns.Current(),
                 nodeAssignsPerInst.Data(), instIds.Current(), nVals);
      sortKeys(level);
    }
  }

  void sortKeys(int level) {
    // segmented-sort the arrays based on node-id's
    // but we don't need more than level+1 bits for sorting!
    SegmentedSort(&tmp_mem, &nodeAssigns, &nodeLocations, nVals, nCols,
                  colOffsets, 0, level + 1);
    dh::Gather<float, int>(dh::GetDeviceIdx(param.gpu_id), vals.other(),
                           vals.Current(), instIds.other(), instIds.Current(),
                           nodeLocations.Current(), nVals);
    vals.buff().selector ^= 1;
    instIds.buff().selector ^= 1;
  }

  void markLeaves() {
    const int BlkDim = 128;
    int nBlks = dh::DivRoundUp(maxNodes, BlkDim);
    markLeavesKernel<<<nBlks, BlkDim>>>(nodes.Data(), maxNodes);
  }
};

XGBOOST_REGISTER_TREE_UPDATER(GPUMaker, "grow_gpu")
    .describe("Grow tree with GPU.")
    .set_body([]() { return new GPUMaker(); });

}  // namespace tree
}  // namespace xgboost
