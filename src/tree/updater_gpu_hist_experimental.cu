#include "hip/hip_runtime.h"
/*!
 * Copyright 2017 XGBoost contributors
 */
#include <thrust/count.h>
#include <thrust/sort.h>
#include <xgboost/tree_updater.h>
#include <algorithm>
#include <memory>
#include <queue>
#include <utility>
#include <vector>
#include "../common/compressed_iterator.h"
#include "../common/device_helpers.cuh"
#include "../common/hist_util.h"
#include "param.h"
#include "updater_gpu_common.cuh"

namespace xgboost {
namespace tree {

DMLC_REGISTRY_FILE_TAG(updater_gpu_hist_experimental);

template <int BLOCK_THREADS, typename reduce_t, typename temp_storage_t>
__device__ bst_gpair_integer ReduceFeature(const bst_gpair_integer* begin,
                                           const bst_gpair_integer* end,
                                           temp_storage_t* temp_storage) {
  __shared__ hipcub::Uninitialized<bst_gpair_integer> uninitialized_sum;
  bst_gpair_integer& shared_sum = uninitialized_sum.Alias();

  bst_gpair_integer local_sum = bst_gpair_integer();
  for (auto itr = begin; itr < end; itr += BLOCK_THREADS) {
    bool thread_active = itr + threadIdx.x < end;
    // Scan histogram
    bst_gpair_integer bin =
        thread_active ? *(itr + threadIdx.x) : bst_gpair_integer();

    local_sum += reduce_t(temp_storage->sum_reduce).Reduce(bin, hipcub::Sum());
  }

  if (threadIdx.x == 0) {
    shared_sum = local_sum;
  }
  __syncthreads();

  return shared_sum;
}

template <int BLOCK_THREADS, typename reduce_t, typename scan_t,
          typename max_reduce_t, typename temp_storage_t>
__device__ void EvaluateFeature(int fidx, const bst_gpair_integer* hist,
                                const int* feature_segments, float min_fvalue,
                                const float* gidx_fvalue_map,
                                DeviceSplitCandidate* best_split,
                                const DeviceNodeStats& node,
                                const GPUTrainingParam& param,
                                temp_storage_t* temp_storage) {
  int gidx_begin = feature_segments[fidx];
  int gidx_end = feature_segments[fidx + 1];

  bst_gpair_integer feature_sum = ReduceFeature<BLOCK_THREADS, reduce_t>(
      hist + gidx_begin, hist + gidx_end, temp_storage);

  auto prefix_op = SumCallbackOp<bst_gpair_integer>();
  for (int scan_begin = gidx_begin; scan_begin < gidx_end;
       scan_begin += BLOCK_THREADS) {
    bool thread_active = scan_begin + threadIdx.x < gidx_end;

    bst_gpair_integer bin =
        thread_active ? hist[scan_begin + threadIdx.x] : bst_gpair_integer();
    scan_t(temp_storage->scan).ExclusiveScan(bin, bin, hipcub::Sum(), prefix_op);

    // Calculate gain
    bst_gpair_integer parent_sum = bst_gpair_integer(node.sum_gradients);

    bst_gpair_integer missing = parent_sum - feature_sum;

    bool missing_left = true;
    const float null_gain = -FLT_MAX;
    float gain = null_gain;
    if (thread_active) {
      gain = loss_chg_missing(bin, missing, parent_sum, node.root_gain, param,
                              missing_left);
    }

    __syncthreads();

    // Find thread with best gain
    hipcub::KeyValuePair<int, float> tuple(threadIdx.x, gain);
    hipcub::KeyValuePair<int, float> best =
        max_reduce_t(temp_storage->max_reduce).Reduce(tuple, hipcub::ArgMax());

    __shared__ hipcub::KeyValuePair<int, float> block_max;
    if (threadIdx.x == 0) {
      block_max = best;
    }

    __syncthreads();

    // Best thread updates split
    if (threadIdx.x == block_max.key) {
      int gidx = scan_begin + threadIdx.x;
      float fvalue =
          gidx == gidx_begin ? min_fvalue : gidx_fvalue_map[gidx - 1];

      bst_gpair_integer left = missing_left ? bin + missing : bin;
      bst_gpair_integer right = parent_sum - left;

      best_split->Update(gain, missing_left ? LeftDir : RightDir, fvalue, fidx,
                         left, right, param);
    }
    __syncthreads();
  }
}

template <int BLOCK_THREADS>
__global__ void evaluate_split_kernel(const bst_gpair_integer* d_hist, int nidx,
                                      int n_features, DeviceNodeStats nodes,
                                      const int* d_feature_segments,
                                      const float* d_fidx_min_map,
                                      const float* d_gidx_fvalue_map,
                                      GPUTrainingParam gpu_param,
                                      DeviceSplitCandidate* d_split) {
  typedef hipcub::KeyValuePair<int, float> ArgMaxT;
  typedef hipcub::BlockScan<bst_gpair_integer, BLOCK_THREADS,
                         hipcub::BLOCK_SCAN_WARP_SCANS>
      BlockScanT;
  typedef hipcub::BlockReduce<ArgMaxT, BLOCK_THREADS> MaxReduceT;

  typedef hipcub::BlockReduce<bst_gpair_integer, BLOCK_THREADS> SumReduceT;

  union TempStorage {
    typename BlockScanT::TempStorage scan;
    typename MaxReduceT::TempStorage max_reduce;
    typename SumReduceT::TempStorage sum_reduce;
  };

  __shared__ hipcub::Uninitialized<DeviceSplitCandidate> uninitialized_split;
  DeviceSplitCandidate& best_split = uninitialized_split.Alias();
  __shared__ TempStorage temp_storage;

  if (threadIdx.x == 0) {
    best_split = DeviceSplitCandidate();
  }

  __syncthreads();

  auto fidx = blockIdx.x;
  EvaluateFeature<BLOCK_THREADS, SumReduceT, BlockScanT, MaxReduceT>(
      fidx, d_hist, d_feature_segments, d_fidx_min_map[fidx], d_gidx_fvalue_map,
      &best_split, nodes, gpu_param, &temp_storage);

  __syncthreads();

  if (threadIdx.x == 0) {
    // Record best loss
    d_split[fidx] = best_split;
  }
}

// Find a gidx value for a given feature otherwise return -1 if not found
template <typename gidx_iter_t>
__device__ int BinarySearchRow(bst_uint begin, bst_uint end, gidx_iter_t data,
                               int fidx_begin, int fidx_end) {
  // for(auto i = begin; i < end; i++)
  //{
  //  auto gidx = data[i];
  //  if (gidx >= fidx_begin&&gidx < fidx_end) return gidx;
  //}
  // return  -1;

  bst_uint previous_middle = UINT32_MAX;
  while (end != begin) {
    auto middle = begin + (end - begin) / 2;
    if (middle == previous_middle) {
      break;
    }
    previous_middle = middle;

    auto gidx = data[middle];

    if (gidx >= fidx_begin && gidx < fidx_end) {
      return gidx;
    } else if (gidx < fidx_begin) {
      begin = middle;
    } else {
      end = middle;
    }
  }
  // Value is missing
  return -1;
}

template <int BLOCK_THREADS>
__global__ void RadixSortSmall(bst_uint* d_ridx, int* d_position, bst_uint n) {
  typedef hipcub::BlockRadixSort<int, BLOCK_THREADS, 1, bst_uint> BlockRadixSort;
  __shared__ typename BlockRadixSort::TempStorage temp_storage;

  bool thread_active = threadIdx.x < n;
  int thread_key[1];
  bst_uint thread_value[1];
  thread_key[0] = thread_active ? d_position[threadIdx.x] : INT_MAX;
  thread_value[0] = thread_active ? d_ridx[threadIdx.x] : UINT_MAX;
  BlockRadixSort(temp_storage).Sort(thread_key, thread_value);

  if (thread_active) {
    d_position[threadIdx.x] = thread_key[0];
    d_ridx[threadIdx.x] = thread_value[0];
  }
}

struct DeviceHistogram {
  dh::bulk_allocator<dh::memory_type::DEVICE> ba;
  dh::dvec<bst_gpair_integer> data;
  std::map<int, bst_gpair_integer*> node_map;
  int n_bins;
  void Init(int device_idx, int max_nodes, int n_bins, bool silent) {
    this->n_bins = n_bins;
    ba.allocate(device_idx, silent, &data, max_nodes * n_bins);
  }

  void Reset() {
    data.fill(bst_gpair_integer());
    node_map.clear();
  }

  void AddNode(int nidx) {
    CHECK_EQ(node_map.count(nidx), 0)
        << nidx << " already exists in the histogram.";
    node_map[nidx] = data.data() + n_bins * node_map.size();
  }
};

// Manage memory for a single GPU
struct DeviceShard {
  int device_idx;
  int normalised_device_idx;  // Device index counting from param.gpu_id
  dh::bulk_allocator<dh::memory_type::DEVICE> ba;
  dh::dvec<common::compressed_byte_t> gidx_buffer;
  dh::dvec<bst_gpair> gpair;
  dh::dvec2<bst_uint> ridx;
  dh::dvec2<int> position;
  std::vector<std::pair<int64_t, int64_t>> ridx_segments;
  dh::dvec<int> feature_segments;
  dh::dvec<float> gidx_fvalue_map;
  dh::dvec<float> min_fvalue;
  std::vector<bst_gpair> node_sum_gradients;
  common::CompressedIterator<uint32_t> gidx;
  int row_stride;
  bst_uint row_start_idx;
  bst_uint row_end_idx;
  bst_uint n_rows;
  int n_bins;
  int null_gidx_value;
  DeviceHistogram hist;

  std::vector<hipStream_t> streams;

  dh::CubMemory temp_memory;

  DeviceShard(int device_idx, int normalised_device_idx,
              const common::GHistIndexMatrix& gmat, bst_uint row_begin,
              bst_uint row_end, int n_bins, TrainParam param)
      : device_idx(device_idx),
        normalised_device_idx(normalised_device_idx),
        row_start_idx(row_begin),
        row_end_idx(row_end),
        n_rows(row_end - row_begin),
        n_bins(n_bins),
        null_gidx_value(n_bins) {
    // Convert to ELLPACK matrix representation
    int max_elements_row = 0;
    for (int i = row_begin; i < row_end; i++) {
      max_elements_row =
          (std::max)(max_elements_row,
                     static_cast<int>(gmat.row_ptr[i + 1] - gmat.row_ptr[i]));
    }
    row_stride = max_elements_row;
    std::vector<int> ellpack_matrix(row_stride * n_rows, null_gidx_value);

    for (int i = row_begin; i < row_end; i++) {
      int row_count = 0;
      for (int j = gmat.row_ptr[i]; j < gmat.row_ptr[i + 1]; j++) {
        ellpack_matrix[i * row_stride + row_count] = gmat.index[j];
        row_count++;
      }
    }

    // Allocate
    int num_symbols = n_bins + 1;
    size_t compressed_size_bytes =
        common::CompressedBufferWriter::CalculateBufferSize(
            ellpack_matrix.size(), num_symbols);
    int max_nodes =
        param.max_leaves > 0 ? param.max_leaves * 2 : n_nodes(param.max_depth);
    ba.allocate(device_idx, param.silent, &gidx_buffer, compressed_size_bytes,
                &gpair, n_rows, &ridx, n_rows, &position, n_rows,
                &feature_segments, gmat.cut->row_ptr.size(), &gidx_fvalue_map,
                gmat.cut->cut.size(), &min_fvalue, gmat.cut->min_val.size());
    gidx_fvalue_map = gmat.cut->cut;
    min_fvalue = gmat.cut->min_val;
    feature_segments = gmat.cut->row_ptr;

    node_sum_gradients.resize(max_nodes);
    ridx_segments.resize(max_nodes);

    // Compress gidx
    common::CompressedBufferWriter cbw(num_symbols);
    std::vector<common::compressed_byte_t> host_buffer(gidx_buffer.size());
    cbw.Write(host_buffer.data(), ellpack_matrix.begin(), ellpack_matrix.end());
    gidx_buffer = host_buffer;
    gidx =
        common::CompressedIterator<uint32_t>(gidx_buffer.data(), num_symbols);

    common::CompressedIterator<uint32_t> ci_host(host_buffer.data(),
                                                 num_symbols);

    // Init histogram
    hist.Init(device_idx, max_nodes, gmat.cut->row_ptr.back(), param.silent);
  }

  ~DeviceShard() {
    for (auto& stream : streams) {
      dh::safe_cuda(hipStreamDestroy(stream));
    }
  }

  // Get vector of at least n initialised streams
  std::vector<hipStream_t>& GetStreams(int n) {
    if (n > streams.size()) {
      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamDestroy(stream));
      }

      streams.clear();
      streams.resize(n);

      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamCreate(&stream));
      }
    }

    return streams;
  }

  // Reset values for each update iteration
  void Reset(const std::vector<bst_gpair>& host_gpair) {
    position.current_dvec().fill(0);
    std::fill(node_sum_gradients.begin(), node_sum_gradients.end(),
              bst_gpair());
    // TODO(rory): support subsampling
    thrust::sequence(ridx.current_dvec().tbegin(), ridx.current_dvec().tend(),
                     row_start_idx);
    std::fill(ridx_segments.begin(), ridx_segments.end(), std::make_pair(0, 0));
    ridx_segments.front() = std::make_pair(0, ridx.size());
    this->gpair.copy(host_gpair.begin() + row_start_idx,
                     host_gpair.begin() + row_end_idx);
    hist.Reset();
  }

  __device__ void IncrementHist(bst_gpair gpair, int gidx,
                                bst_gpair_integer* node_hist) const {
    auto dst_ptr =
        reinterpret_cast<unsigned long long int*>(&node_hist[gidx]);  // NOLINT
    bst_gpair_integer tmp(gpair.GetGrad(), gpair.GetHess());
    auto src_ptr = reinterpret_cast<bst_gpair_integer::value_t*>(&tmp);

    atomicAdd(dst_ptr,
              static_cast<unsigned long long int>(*src_ptr));  // NOLINT
    atomicAdd(dst_ptr + 1,
              static_cast<unsigned long long int>(*(src_ptr + 1)));  // NOLINT
  }

  void BuildHist(int nidx) {
    hist.AddNode(nidx);
    auto d_node_hist = hist.node_map[nidx];
    auto d_gidx = gidx;
    auto d_ridx = ridx.current();
    auto d_gpair = gpair.data();
    auto row_stride = this->row_stride;
    auto null_gidx_value = this->null_gidx_value;
    auto segment = ridx_segments[nidx];
    auto n_elements = (segment.second - segment.first) * row_stride;

    dh::launch_n(device_idx, n_elements, [=] __device__(size_t idx) {
      int relative_ridx = d_ridx[(idx / row_stride) + segment.first];
      int gidx = d_gidx[relative_ridx * row_stride + idx % row_stride];
      if (gidx != null_gidx_value) {
        bst_gpair gpair = d_gpair[relative_ridx];
        IncrementHist(gpair, gidx, d_node_hist);
      }
    });
  }
  void SortPosition(const std::pair<bst_uint, bst_uint>& segment, int left_nidx,
                    int right_nidx) {
    auto n = segment.second - segment.first;
    int min_bits = 0;
    int max_bits = std::ceil(std::log2((std::max)(left_nidx, right_nidx) + 1));
    // const int SINGLE_TILE_SIZE = 1024;
    // if (n < SINGLE_TILE_SIZE) {
    //  RadixSortSmall<SINGLE_TILE_SIZE>
    //      <<<1, SINGLE_TILE_SIZE>>>(ridx.current() + segment.first,
    //                                position.current() + segment.first, n);
    //} else {

    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(
        nullptr, temp_storage_bytes, position.current() + segment.first,
        position.other() + segment.first, ridx.current() + segment.first,
        ridx.other() + segment.first, n, min_bits, max_bits);

    temp_memory.LazyAllocate(temp_storage_bytes);

    hipcub::DeviceRadixSort::SortPairs(
        temp_memory.d_temp_storage, temp_memory.temp_storage_bytes,
        position.current() + segment.first, position.other() + segment.first,
        ridx.current() + segment.first, ridx.other() + segment.first, n,
        min_bits, max_bits);
    dh::safe_cuda(hipMemcpy(position.current() + segment.first,
                             position.other() + segment.first, n * sizeof(int),
                             hipMemcpyDeviceToDevice));
    dh::safe_cuda(hipMemcpy(ridx.current() + segment.first,
                             ridx.other() + segment.first, n * sizeof(bst_uint),
                             hipMemcpyDeviceToDevice));
    //}
  }
};

class GPUHistMakerExperimental : public TreeUpdater {
 public:
  struct ExpandEntry;

  GPUHistMakerExperimental() : initialised(false) {}
  ~GPUHistMakerExperimental() {}
  void Init(
      const std::vector<std::pair<std::string, std::string>>& args) override {
    param.InitAllowUnknown(args);
    CHECK(param.n_gpus != 0) << "Must have at least one device";
    CHECK(param.n_gpus <= 1 && param.n_gpus != -1)
        << "Only one GPU currently supported";
    n_devices = param.n_gpus;

    if (param.grow_policy == TrainParam::kLossGuide) {
      qexpand_.reset(new ExpandQueue(loss_guide));
    } else {
      qexpand_.reset(new ExpandQueue(depth_wise));
    }

    monitor.Init("updater_gpu_hist_experimental", param.debug_verbose);
  }
  void Update(const std::vector<bst_gpair>& gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) override {
    GradStats::CheckInfo(dmat->info());
    // rescale learning rate according to size of trees
    float lr = param.learning_rate;
    param.learning_rate = lr / trees.size();
    // build tree
    try {
      for (size_t i = 0; i < trees.size(); ++i) {
        this->UpdateTree(gpair, dmat, trees[i]);
      }
    } catch (const std::exception& e) {
      LOG(FATAL) << "GPU plugin exception: " << e.what() << std::endl;
    }
    param.learning_rate = lr;
  }

  void InitDataOnce(DMatrix* dmat) {
    info = &dmat->info();
    hmat_.Init(dmat, param.max_bin);
    gmat_.cut = &hmat_;
    gmat_.Init(dmat);
    n_bins = hmat_.row_ptr.back();
    shards.emplace_back(param.gpu_id, 0, gmat_, 0, info->num_row, n_bins,
                        param);
    initialised = true;
  }

  void InitData(const std::vector<bst_gpair>& gpair, DMatrix* dmat,
                const RegTree& tree) {
    if (!initialised) {
      this->InitDataOnce(dmat);
    }

    this->ColSampleTree();

    // Copy gpair & reset memory
    for (auto& shard : shards) {
      shard.Reset(gpair);
    }
  }

  void BuildHist(int nidx) {
    for (auto& shard : shards) {
      shard.BuildHist(nidx);
    }
  }

  // Returns best loss
  std::vector<DeviceSplitCandidate> EvaluateSplits(
      const std::vector<int>& nidx_set, RegTree* p_tree) {
    auto columns = info->num_col;
    std::vector<DeviceSplitCandidate> best_splits(nidx_set.size());
    std::vector<DeviceSplitCandidate> candidate_splits(nidx_set.size() *
                                                       columns);
    // Use first device
    auto& shard = shards.front();
    dh::safe_cuda(hipSetDevice(shard.device_idx));
    shard.temp_memory.LazyAllocate(sizeof(DeviceSplitCandidate) * columns *
                                   nidx_set.size());
    auto d_split = shard.temp_memory.Pointer<DeviceSplitCandidate>();

    auto& streams = shard.GetStreams(nidx_set.size());

    // Use streams to process nodes concurrently
    for (auto i = 0; i < nidx_set.size(); i++) {
      auto nidx = nidx_set[i];
      DeviceNodeStats node(shard.node_sum_gradients[nidx], nidx, param);

      const int BLOCK_THREADS = 256;
      evaluate_split_kernel<BLOCK_THREADS>
          <<<columns, BLOCK_THREADS, 0, streams[i]>>>(
              shard.hist.node_map[nidx], nidx, info->num_col, node,
              shard.feature_segments.data(), shard.min_fvalue.data(),
              shard.gidx_fvalue_map.data(), GPUTrainingParam(param),
              d_split + i * columns);
    }

    dh::safe_cuda(
        hipMemcpy(candidate_splits.data(), shard.temp_memory.d_temp_storage,
                   sizeof(DeviceSplitCandidate) * columns * nidx_set.size(),
                   hipMemcpyDeviceToHost));

    for (auto i = 0; i < nidx_set.size(); i++) {
      DeviceSplitCandidate nidx_best;
      for (auto fidx = 0; fidx < columns; fidx++) {
        nidx_best.Update(candidate_splits[i * columns + fidx], param);
      }
      best_splits[i] = nidx_best;
    }
    return std::move(best_splits);
  }

  void InitRoot(const std::vector<bst_gpair>& gpair, RegTree* p_tree) {
    int root_nidx = 0;
    BuildHist(root_nidx);

    // TODO(rory): support sub sampling
    // TODO(rory): not asynchronous
    bst_gpair sum_gradient;
    for (auto& shard : shards) {
      sum_gradient += thrust::reduce(shard.gpair.tbegin(), shard.gpair.tend());
    }

    // Remember root stats
    p_tree->stat(root_nidx).sum_hess = sum_gradient.GetHess();
    p_tree->stat(root_nidx).base_weight = CalcWeight(param, sum_gradient);

    // Store sum gradients
    for (auto& shard : shards) {
      shard.node_sum_gradients[root_nidx] = sum_gradient;
    }

    auto splits = this->EvaluateSplits({root_nidx}, p_tree);

    // Generate candidate
    qexpand_->push(
        ExpandEntry(root_nidx, p_tree->GetDepth(root_nidx), splits.front(), 0));
  }

  struct MatchingFunctor : public thrust::unary_function<int, int> {
    int val;
    __host__ __device__ MatchingFunctor(int val) : val(val) {}
    __host__ __device__ int operator()(int x) const { return x == val; }
  };

  __device__ void CountLeft(bst_uint* d_count, int val, int left_nidx) {
    unsigned ballot = __ballot(val == left_nidx);
    if (threadIdx.x % 32 == 0) {
      atomicAdd(d_count, __popc(ballot));
    }
  }

  void UpdatePosition(const ExpandEntry& candidate, RegTree* p_tree) {
    auto nidx = candidate.nid;
    auto is_dense = info->num_nonzero == info->num_row * info->num_col;
    auto left_nidx = (*p_tree)[nidx].cleft();
    auto right_nidx = (*p_tree)[nidx].cright();

    // convert floating-point split_pt into corresponding bin_id
    // split_cond = -1 indicates that split_pt is less than all known cut points
    auto split_gidx = -1;
    auto fidx = candidate.split.findex;
    auto default_dir_left = candidate.split.dir == LeftDir;
    auto fidx_begin = hmat_.row_ptr[fidx];
    auto fidx_end = hmat_.row_ptr[fidx + 1];
    for (auto i = fidx_begin; i < fidx_end; ++i) {
      if (candidate.split.fvalue == hmat_.cut[i]) {
        split_gidx = static_cast<int32_t>(i);
      }
    }

    for (auto& shard : shards) {
      monitor.Start("update position kernel");
      shard.temp_memory.LazyAllocate(sizeof(bst_uint));
      auto d_left_count = shard.temp_memory.Pointer<bst_uint>();
      dh::safe_cuda(hipMemset(d_left_count, 0, sizeof(bst_uint)));
      dh::safe_cuda(hipSetDevice(shard.device_idx));
      auto segment = shard.ridx_segments[nidx];
      CHECK_GT(segment.second - segment.first, 0);
      auto d_ridx = shard.ridx.current();
      auto d_position = shard.position.current();
      auto d_gidx = shard.gidx;
      auto row_stride = shard.row_stride;
      dh::launch_n<1, 512>(
          shard.device_idx, segment.second - segment.first,
          [=] __device__(bst_uint idx) {
            idx += segment.first;
            auto ridx = d_ridx[idx];
            auto row_begin = row_stride * ridx;
            auto row_end = row_begin + row_stride;
            auto gidx = -1;
            if (is_dense) {
              gidx = d_gidx[row_begin + fidx];
            } else {
              gidx = BinarySearchRow(row_begin, row_end, d_gidx, fidx_begin,
                                     fidx_end);
            }

            int position;
            if (gidx >= 0) {
              // Feature is found
              position = gidx <= split_gidx ? left_nidx : right_nidx;
            } else {
              // Feature is missing
              position = default_dir_left ? left_nidx : right_nidx;
            }

            CountLeft(d_left_count, position, left_nidx);
            d_position[idx] = position;
          });

      bst_uint left_count;
      dh::safe_cuda(hipMemcpy(&left_count, d_left_count, sizeof(bst_uint),
                               hipMemcpyDeviceToHost));
      monitor.Stop("update position kernel");

      monitor.Start("sort");
      shard.SortPosition(segment, left_nidx, right_nidx);
      monitor.Stop("sort");
      shard.ridx_segments[left_nidx] =
          std::make_pair(segment.first, segment.first + left_count);
      shard.ridx_segments[right_nidx] =
          std::make_pair(segment.first + left_count, segment.second);
    }
  }

  void ApplySplit(const ExpandEntry& candidate, RegTree* p_tree) {
    // Add new leaves
    RegTree& tree = *p_tree;
    tree.AddChilds(candidate.nid);
    auto& parent = tree[candidate.nid];
    parent.set_split(candidate.split.findex, candidate.split.fvalue,
                     candidate.split.dir == LeftDir);
    tree.stat(candidate.nid).loss_chg = candidate.split.loss_chg;

    // Configure left child
    auto left_weight = CalcWeight(param, candidate.split.left_sum);
    tree[parent.cleft()].set_leaf(left_weight * param.learning_rate, 0);
    tree.stat(parent.cleft()).base_weight = left_weight;
    tree.stat(parent.cleft()).sum_hess = candidate.split.left_sum.GetHess();

    // Configure right child
    auto right_weight = CalcWeight(param, candidate.split.right_sum);
    tree[parent.cright()].set_leaf(right_weight * param.learning_rate, 0);
    tree.stat(parent.cright()).base_weight = right_weight;
    tree.stat(parent.cright()).sum_hess = candidate.split.right_sum.GetHess();
    // Store sum gradients
    for (auto& shard : shards) {
      shard.node_sum_gradients[parent.cleft()] = candidate.split.left_sum;
      shard.node_sum_gradients[parent.cright()] = candidate.split.right_sum;
    }
    this->UpdatePosition(candidate, p_tree);
  }

  void ColSampleTree() {
    if (param.colsample_bylevel == 1.0 && param.colsample_bytree == 1.0) return;

    feature_set_tree.resize(info->num_col);
    std::iota(feature_set_tree.begin(), feature_set_tree.end(), 0);
    feature_set_tree = col_sample(feature_set_tree, param.colsample_bytree);
  }

  struct Monitor {
    bool debug_verbose = false;
    std::string label = "";
    std::map<std::string, dh::Timer> timer_map;

    ~Monitor() {
      if (!debug_verbose) return;

      std::cout << "Monitor: " << label << "\n";
      for (auto& kv : timer_map) {
        kv.second.PrintElapsed(kv.first);
      }
    }
    void Init(std::string label, bool debug_verbose) {
      this->debug_verbose = debug_verbose;
      this->label = label;
    }
    void Start(const std::string& name) { timer_map[name].Start(); }
    void Stop(const std::string& name) { timer_map[name].Stop(); }
  };

  void UpdateTree(const std::vector<bst_gpair>& gpair, DMatrix* p_fmat,
                  RegTree* p_tree) {
    auto& tree = *p_tree;

    monitor.Start("InitData");
    this->InitData(gpair, p_fmat, *p_tree);
    monitor.Stop("InitData");
    monitor.Start("InitRoot");
    this->InitRoot(gpair, p_tree);
    monitor.Stop("InitRoot");

    unsigned timestamp = qexpand_->size();
    auto num_leaves = 1;

    while (!qexpand_->empty()) {
      auto candidate = qexpand_->top();
      qexpand_->pop();
      if (!candidate.IsValid(param, num_leaves)) continue;
      // std::cout << candidate;
      monitor.Start("ApplySplit");
      this->ApplySplit(candidate, p_tree);
      monitor.Stop("ApplySplit");
      num_leaves++;

      auto left_child_nidx = tree[candidate.nid].cleft();
      auto right_child_nidx = tree[candidate.nid].cright();

      // Only create child entries if needed
      if (ExpandEntry::ChildIsValid(param, tree.GetDepth(left_child_nidx),
                                    num_leaves)) {
        monitor.Start("BuildHist");
        this->BuildHist(left_child_nidx);
        this->BuildHist(right_child_nidx);
        monitor.Stop("BuildHist");

        monitor.Start("EvaluateSplits");
        auto splits =
            this->EvaluateSplits({left_child_nidx, right_child_nidx}, p_tree);
        qexpand_->push(ExpandEntry(left_child_nidx,
                                   tree.GetDepth(left_child_nidx), splits[0],
                                   timestamp++));
        qexpand_->push(ExpandEntry(right_child_nidx,
                                   tree.GetDepth(right_child_nidx), splits[1],
                                   timestamp++));
        monitor.Stop("EvaluateSplits");
      }
    }
  }

  struct ExpandEntry {
    int nid;
    int depth;
    DeviceSplitCandidate split;
    unsigned timestamp;
    ExpandEntry(int nid, int depth, const DeviceSplitCandidate& split,
                unsigned timestamp)
        : nid(nid), depth(depth), split(split), timestamp(timestamp) {}
    bool IsValid(const TrainParam& param, int num_leaves) const {
      if (split.loss_chg <= rt_eps) return false;
      if (param.max_depth > 0 && depth == param.max_depth) return false;
      if (param.max_leaves > 0 && num_leaves == param.max_leaves) return false;
      return true;
    }

    static bool ChildIsValid(const TrainParam& param, int depth,
                             int num_leaves) {
      if (param.max_depth > 0 && depth == param.max_depth) return false;
      if (param.max_leaves > 0 && num_leaves == param.max_leaves) return false;
      return true;
    }

    friend std::ostream& operator<<(std::ostream& os, const ExpandEntry& e) {
      os << "ExpandEntry: \n";
      os << "nidx: " << e.nid << "\n";
      os << "depth: " << e.depth << "\n";
      os << "loss: " << e.split.loss_chg << "\n";
      os << "left_sum: " << e.split.left_sum << "\n";
      os << "right_sum: " << e.split.right_sum << "\n";
      return os;
    }
  };

  inline static bool depth_wise(ExpandEntry lhs, ExpandEntry rhs) {
    if (lhs.depth == rhs.depth) {
      return lhs.timestamp > rhs.timestamp;  // favor small timestamp
    } else {
      return lhs.depth > rhs.depth;  // favor small depth
    }
  }
  inline static bool loss_guide(ExpandEntry lhs, ExpandEntry rhs) {
    if (lhs.split.loss_chg == rhs.split.loss_chg) {
      return lhs.timestamp > rhs.timestamp;  // favor small timestamp
    } else {
      return lhs.split.loss_chg < rhs.split.loss_chg;  // favor large loss_chg
    }
  }
  TrainParam param;
  common::HistCutMatrix hmat_;
  common::GHistIndexMatrix gmat_;
  MetaInfo* info;
  bool initialised;
  int n_devices;
  int n_bins;

  std::vector<DeviceShard> shards;
  std::vector<int> feature_set_tree;
  std::vector<int> feature_set_level;
  typedef std::priority_queue<ExpandEntry, std::vector<ExpandEntry>,
                              std::function<bool(ExpandEntry, ExpandEntry)>>
      ExpandQueue;
  std::unique_ptr<ExpandQueue> qexpand_;
  Monitor monitor;
};

XGBOOST_REGISTER_TREE_UPDATER(GPUHistMakerExperimental,
                              "grow_gpu_hist_experimental")
    .describe("Grow tree with GPU.")
    .set_body([]() { return new GPUHistMakerExperimental(); });
}  // namespace tree
}  // namespace xgboost
