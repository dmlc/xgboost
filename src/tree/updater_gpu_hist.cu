#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2018 XGBoost contributors
 */
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <xgboost/tree_updater.h>
#include <algorithm>
#include <cmath>
#include <memory>
#include <limits>
#include <queue>
#include <utility>
#include <vector>
#include "../common/common.h"
#include "../common/compressed_iterator.h"
#include "../common/device_helpers.cuh"
#include "../common/hist_util.h"
#include "../common/host_device_vector.h"
#include "../common/timer.h"
#include "../common/span.h"
#include "param.h"
#include "updater_gpu_common.cuh"

namespace xgboost {
namespace tree {

DMLC_REGISTRY_FILE_TAG(updater_gpu_hist);

// training parameters specific to this algorithm
struct GPUHistMakerTrainParam
    : public dmlc::Parameter<GPUHistMakerTrainParam> {
  bool single_precision_histogram;
  // number of rows in a single GPU batch
  int gpu_batch_nrows;
  // declare parameters
  DMLC_DECLARE_PARAMETER(GPUHistMakerTrainParam) {
    DMLC_DECLARE_FIELD(single_precision_histogram).set_default(false).describe(
        "Use single precision to build histograms.");
    DMLC_DECLARE_FIELD(gpu_batch_nrows)
        .set_lower_bound(-1)
        .set_default(0)
        .describe("Number of rows in a GPU batch, used for finding quantiles on GPU; "
                  "-1 to use all rows assignted to a GPU, and 0 to auto-deduce");
  }
};

DMLC_REGISTER_PARAMETER(GPUHistMakerTrainParam);

/*!
 * \brief
 *
 * \tparam ReduceT     BlockReduce Type.
 * \tparam TempStorage Cub Shared memory
 *
 * \param begin
 * \param end
 * \param temp_storage Shared memory for intermediate result.
 */
template <int BLOCK_THREADS, typename ReduceT, typename TempStorageT, typename GradientSumT>
__device__ GradientSumT ReduceFeature(common::Span<const GradientSumT> feature_histogram,
                                      TempStorageT* temp_storage) {
  __shared__ hipcub::Uninitialized<GradientSumT> uninitialized_sum;
  GradientSumT& shared_sum = uninitialized_sum.Alias();

  GradientSumT local_sum = GradientSumT();
  // For loop sums features into one block size
  auto begin = feature_histogram.data();
  auto end = begin + feature_histogram.size();
  for (auto itr = begin; itr < end; itr += BLOCK_THREADS) {
    bool thread_active = itr + threadIdx.x < end;
    // Scan histogram
    GradientSumT bin = thread_active ? *(itr + threadIdx.x) : GradientSumT();
    local_sum += bin;
  }
  local_sum = ReduceT(temp_storage->sum_reduce).Reduce(local_sum, hipcub::Sum());
  // Reduction result is stored in thread 0.
  if (threadIdx.x == 0) {
    shared_sum = local_sum;
  }
  __syncthreads();
  return shared_sum;
}

/*! \brief Find the thread with best gain. */
template <int BLOCK_THREADS, typename ReduceT, typename scan_t,
          typename MaxReduceT, typename TempStorageT, typename GradientSumT>
__device__ void EvaluateFeature(
    int fidx,
    common::Span<const GradientSumT> node_histogram,
    common::Span<const uint32_t> feature_segments,  // cut.row_ptr
    float min_fvalue,                               // cut.min_value
    common::Span<const float> gidx_fvalue_map,                   // cut.cut
    DeviceSplitCandidate* best_split,  // shared memory storing best split
    const DeviceNodeStats& node, const GPUTrainingParam& param,
    TempStorageT* temp_storage,  // temp memory for cub operations
    int constraint,              // monotonic_constraints
    const ValueConstraint& value_constraint) {
  // Use pointer from cut to indicate begin and end of bins for each feature.
  uint32_t gidx_begin = feature_segments[fidx];    // begining bin
  uint32_t gidx_end = feature_segments[fidx + 1];  // end bin for i^th feature

  // Sum histogram bins for current feature
  GradientSumT const feature_sum = ReduceFeature<BLOCK_THREADS, ReduceT>(
      node_histogram.subspan(gidx_begin, gidx_end - gidx_begin), temp_storage);

  GradientSumT const parent_sum = GradientSumT(node.sum_gradients);
  GradientSumT const missing = parent_sum - feature_sum;
  float const null_gain = -std::numeric_limits<bst_float>::infinity();

  SumCallbackOp<GradientSumT> prefix_op =
      SumCallbackOp<GradientSumT>();
  for (int scan_begin = gidx_begin; scan_begin < gidx_end;
       scan_begin += BLOCK_THREADS) {
    bool thread_active = (scan_begin + threadIdx.x) < gidx_end;

    // Gradient value for current bin.
    GradientSumT bin =
        thread_active ? node_histogram[scan_begin + threadIdx.x] : GradientSumT();
    scan_t(temp_storage->scan).ExclusiveScan(bin, bin, hipcub::Sum(), prefix_op);

    // Whether the gradient of missing values is put to the left side.
    bool missing_left = true;
    float gain = null_gain;
    if (thread_active) {
      gain = LossChangeMissing(bin, missing, parent_sum, node.root_gain, param,
                               constraint, value_constraint, missing_left);
    }

    __syncthreads();

    // Find thread with best gain
    hipcub::KeyValuePair<int, float> tuple(threadIdx.x, gain);
    hipcub::KeyValuePair<int, float> best =
        MaxReduceT(temp_storage->max_reduce).Reduce(tuple, hipcub::ArgMax());

    __shared__ hipcub::KeyValuePair<int, float> block_max;
    if (threadIdx.x == 0) {
      block_max = best;
    }

    __syncthreads();

    // Best thread updates split
    if (threadIdx.x == block_max.key) {
      int gidx = scan_begin + threadIdx.x;
      float fvalue =
          gidx == gidx_begin ? min_fvalue : gidx_fvalue_map[gidx - 1];
      GradientSumT left = missing_left ? bin + missing : bin;
      GradientSumT right = parent_sum - left;
      best_split->Update(gain, missing_left ? kLeftDir : kRightDir,
                         fvalue, fidx,
                         GradientPair(left),
                         GradientPair(right),
                         param);
    }
    __syncthreads();
  }
}

template <int BLOCK_THREADS, typename GradientSumT>
__global__ void EvaluateSplitKernel(
    common::Span<const GradientSumT>
        node_histogram,               // histogram for gradients
    common::Span<const int> feature_set,  // Selected features
    DeviceNodeStats node,
    common::Span<const uint32_t>
        d_feature_segments,                       // row_ptr form HistCutMatrix
    common::Span<const float> d_fidx_min_map,     // min_value
    common::Span<const float> d_gidx_fvalue_map,  // cut
    GPUTrainingParam gpu_param,
    common::Span<DeviceSplitCandidate> split_candidates,  // resulting split
    ValueConstraint value_constraint,
    common::Span<int> d_monotonic_constraints) {
  // KeyValuePair here used as threadIdx.x -> gain_value
  typedef hipcub::KeyValuePair<int, float> ArgMaxT;
  typedef hipcub::BlockScan<
    GradientSumT, BLOCK_THREADS, hipcub::BLOCK_SCAN_WARP_SCANS> BlockScanT;
  typedef hipcub::BlockReduce<ArgMaxT, BLOCK_THREADS> MaxReduceT;

  typedef hipcub::BlockReduce<GradientSumT, BLOCK_THREADS> SumReduceT;

  union TempStorage {
    typename BlockScanT::TempStorage scan;
    typename MaxReduceT::TempStorage max_reduce;
    typename SumReduceT::TempStorage sum_reduce;
  };

  // Aligned && shared storage for best_split
  __shared__ hipcub::Uninitialized<DeviceSplitCandidate> uninitialized_split;
  DeviceSplitCandidate& best_split = uninitialized_split.Alias();
  __shared__ TempStorage temp_storage;

  if (threadIdx.x == 0) {
    best_split = DeviceSplitCandidate();
  }

  __syncthreads();

  // One block for each feature. Features are sampled, so fidx != blockIdx.x
  int fidx = feature_set[blockIdx.x];
  int constraint = d_monotonic_constraints[fidx];
  EvaluateFeature<BLOCK_THREADS, SumReduceT, BlockScanT, MaxReduceT>(
      fidx, node_histogram,
      d_feature_segments, d_fidx_min_map[fidx], d_gidx_fvalue_map,
      &best_split, node, gpu_param, &temp_storage, constraint,
      value_constraint);

  __syncthreads();

  if (threadIdx.x == 0) {
    // Record best loss for each feature
    split_candidates[blockIdx.x] = best_split;
  }
}

// Find a gidx value for a given feature otherwise return -1 if not found
template <typename GidxIterT>
__device__ int BinarySearchRow(bst_uint begin, bst_uint end, GidxIterT data,
                               int const fidx_begin, int const fidx_end) {
  bst_uint previous_middle = UINT32_MAX;
  while (end != begin) {
    auto middle = begin + (end - begin) / 2;
    if (middle == previous_middle) {
      break;
    }
    previous_middle = middle;

    auto gidx = data[middle];

    if (gidx >= fidx_begin && gidx < fidx_end) {
      return gidx;
    } else if (gidx < fidx_begin) {
      begin = middle;
    } else {
      end = middle;
    }
  }
  // Value is missing
  return -1;
}

/**
 * \struct  DeviceHistogram
 *
 * \summary Data storage for node histograms on device. Automatically expands.
 *
 * \author  Rory
 * \date    28/07/2018
 */
template <typename GradientSumT>
struct DeviceHistogram {
  /*! \brief Map nidx to starting index of its histogram. */
  std::map<int, size_t> nidx_map;
  thrust::device_vector<typename GradientSumT::ValueT> data;
  const size_t kStopGrowingSize = 1 << 26;  // Do not grow beyond this size
  int n_bins;
  int device_id_;

  void Init(int device_id, int n_bins) {
    this->n_bins = n_bins;
    this->device_id_ = device_id;
  }

  void Reset() {
    dh::safe_cuda(hipSetDevice(device_id_));
    data.resize(0);
    nidx_map.clear();
  }

  bool HistogramExists(int nidx) {
    return nidx_map.find(nidx) != nidx_map.end();
  }

  void AllocateHistogram(int nidx) {
    if (HistogramExists(nidx)) return;

    if (data.size() > kStopGrowingSize) {
      // Recycle histogram memory
      std::pair<int, size_t> old_entry = *nidx_map.begin();
      nidx_map.erase(old_entry.first);
      dh::safe_cuda(hipMemset(data.data().get() + old_entry.second, 0,
                               n_bins * sizeof(GradientSumT)));
      nidx_map[nidx] = old_entry.second;
    } else {
      // Append new node histogram
      nidx_map[nidx] = data.size();
      dh::safe_cuda(hipSetDevice(device_id_));
      // x 2: Hess and Grad.
      data.resize(data.size() + (n_bins * 2));
    }
  }

  /**
   * \summary   Return pointer to histogram memory for a given node.
   * \param nidx    Tree node index.
   * \return    hist pointer.
   */
  common::Span<GradientSumT> GetNodeHistogram(int nidx) {
    CHECK(this->HistogramExists(nidx));
    auto ptr = data.data().get() + nidx_map[nidx];
    return common::Span<GradientSumT>(
        reinterpret_cast<GradientSumT*>(ptr), n_bins);
  }
};

struct CalcWeightTrainParam {
  float min_child_weight;
  float reg_alpha;
  float reg_lambda;
  float max_delta_step;
  float learning_rate;
  XGBOOST_DEVICE explicit CalcWeightTrainParam(const TrainParam& p)
      : min_child_weight(p.min_child_weight),
        reg_alpha(p.reg_alpha),
        reg_lambda(p.reg_lambda),
        max_delta_step(p.max_delta_step),
        learning_rate(p.learning_rate) {}
};

// Bin each input data entry, store the bin indices in compressed form.
__global__ void compress_bin_ellpack_k(
    common::CompressedBufferWriter wr,
    common::CompressedByteT* __restrict__ buffer,  // gidx_buffer
    const size_t* __restrict__ row_ptrs,           // row offset of input data
    const Entry* __restrict__ entries,      // One batch of input data
    const float* __restrict__ cuts,         // HistCutMatrix::cut
    const uint32_t* __restrict__ cut_rows,  // HistCutMatrix::row_ptrs
    size_t base_row,                        // batch_row_begin
    size_t n_rows,
    // row_ptr_begin: row_offset[base_row], the start position of base_row
    size_t row_ptr_begin,
    size_t row_stride,
    unsigned int null_gidx_value) {
  size_t irow = threadIdx.x + blockIdx.x * blockDim.x;
  int ifeature = threadIdx.y + blockIdx.y * blockDim.y;
  if (irow >= n_rows || ifeature >= row_stride)
    return;
  int row_length = static_cast<int>(row_ptrs[irow + 1] - row_ptrs[irow]);
  unsigned int bin = null_gidx_value;
  if (ifeature < row_length) {
    Entry entry = entries[row_ptrs[irow] - row_ptr_begin + ifeature];
    int feature = entry.index;
    float fvalue = entry.fvalue;
    // {feature_cuts, ncuts} forms the array of cuts of `feature'.
    const float *feature_cuts = &cuts[cut_rows[feature]];
    int ncuts = cut_rows[feature + 1] - cut_rows[feature];
    // Assigning the bin in current entry.
    // S.t.: fvalue < feature_cuts[bin]
    bin = dh::UpperBound(feature_cuts, ncuts, fvalue);
    if (bin >= ncuts)
      bin = ncuts - 1;
    // Add the number of bins in previous features.
    bin += cut_rows[feature];
  }
  // Write to gidx buffer.
  wr.AtomicWriteSymbol(buffer, bin, (irow + base_row) * row_stride + ifeature);
}

template <typename GradientSumT>
__global__ void SharedMemHistKernel(size_t row_stride, const bst_uint* d_ridx,
                                    common::CompressedIterator<uint32_t> d_gidx,
                                    int null_gidx_value,
                                    GradientSumT* d_node_hist,
                                    const GradientPair* d_gpair,
                                    size_t segment_begin, size_t n_elements) {
  extern __shared__ char smem[];
  GradientSumT* smem_arr = reinterpret_cast<GradientSumT*>(smem); // NOLINT
  for (auto i : dh::BlockStrideRange(0, null_gidx_value)) {
    smem_arr[i] = GradientSumT();
  }
  __syncthreads();
  for (auto idx : dh::GridStrideRange(static_cast<size_t>(0), n_elements)) {
    int ridx = d_ridx[idx / row_stride + segment_begin];
    int gidx = d_gidx[ridx * row_stride + idx % row_stride];
    if (gidx != null_gidx_value) {
      AtomicAddGpair(smem_arr + gidx, d_gpair[ridx]);
    }
  }
  __syncthreads();
  for (auto i : dh::BlockStrideRange(0, null_gidx_value)) {
    AtomicAddGpair(d_node_hist + i, smem_arr[i]);
  }
}

struct Segment {
  size_t begin;
  size_t end;

  Segment() : begin(0), end(0) {}

  Segment(size_t begin, size_t end) : begin(begin), end(end) {
    CHECK_GE(end, begin);
  }
  size_t Size() const { return end - begin; }
};

/** \brief Returns a one if the left node index is encountered, otherwise return
 * zero. */
struct IndicateLeftTransform {
  int left_nidx;
  explicit IndicateLeftTransform(int left_nidx) : left_nidx(left_nidx) {}
  __host__ __device__ __forceinline__ int operator()(const int& x) const {
    return x == left_nidx ? 1 : 0;
  }
};

/**
 * \brief Optimised routine for sorting key value pairs into left and right
 * segments. Based on a single pass of exclusive scan, uses iterators to
 * redirect inputs and outputs.
 */
void SortPosition(dh::CubMemory* temp_memory, common::Span<int> position,
                  common::Span<int> position_out, common::Span<bst_uint> ridx,
                  common::Span<bst_uint> ridx_out, int left_nidx,
                  int right_nidx, int64_t left_count) {
  auto d_position_out = position_out.data();
  auto d_position_in = position.data();
  auto d_ridx_out = ridx_out.data();
  auto d_ridx_in = ridx.data();
  auto write_results = [=] __device__(size_t idx, int ex_scan_result) {
    int scatter_address;
    if (d_position_in[idx] == left_nidx) {
      scatter_address = ex_scan_result;
    } else {
      scatter_address = (idx - ex_scan_result) + left_count;
    }
    d_position_out[scatter_address] = d_position_in[idx];
    d_ridx_out[scatter_address] = d_ridx_in[idx];
  };  // NOLINT

  IndicateLeftTransform conversion_op(left_nidx);
  hipcub::TransformInputIterator<int, IndicateLeftTransform, int*> in_itr(
      d_position_in, conversion_op);
  dh::DiscardLambdaItr<decltype(write_results)> out_itr(write_results);
  size_t temp_storage_bytes = 0;
  hipcub::DeviceScan::ExclusiveSum(nullptr, temp_storage_bytes, in_itr, out_itr,
                                position.size());
  temp_memory->LazyAllocate(temp_storage_bytes);
  hipcub::DeviceScan::ExclusiveSum(temp_memory->d_temp_storage,
                                temp_memory->temp_storage_bytes, in_itr,
                                out_itr, position.size());
}

template <typename GradientSumT>
struct DeviceShard;

template <typename GradientSumT>
struct GPUHistBuilderBase {
 public:
  virtual void Build(DeviceShard<GradientSumT>* shard, int idx) = 0;
  virtual ~GPUHistBuilderBase() = default;
};

// Manage memory for a single GPU
template <typename GradientSumT>
struct DeviceShard {
  int device_id_;
  dh::BulkAllocator<dh::MemoryType::kDevice> ba;

  /*! \brief HistCutMatrix stored in device. */
  struct DeviceHistCutMatrix {
    /*! \brief row_ptr form HistCutMatrix. */
    dh::DVec<uint32_t> feature_segments;
    /*! \brief minimum value for each feature. */
    dh::DVec<bst_float> min_fvalue;
    /*! \brief Cut. */
    dh::DVec<bst_float> gidx_fvalue_map;
  } cut_;

  /*! \brief Range of rows for each node. */
  std::vector<Segment> ridx_segments;
  DeviceHistogram<GradientSumT> hist;

  /*! \brief global index of histogram, which is stored in ELLPack format. */
  dh::DVec<common::CompressedByteT> gidx_buffer;
  /*! \brief row length for ELLPack. */
  size_t row_stride;
  common::CompressedIterator<uint32_t> gidx;

  /*! \brief  Row indices relative to this shard, necessary for sorting rows. */
  dh::DVec2<bst_uint> ridx;
  /*! \brief Gradient pair for each row. */
  dh::DVec<GradientPair> gpair;

  /*! \brief The last histogram index. */
  int null_gidx_value;

  dh::DVec2<int> position;

  dh::DVec<int> monotone_constraints;
  dh::DVec<bst_float> prediction_cache;

  /*! \brief Sum gradient for each node. */
  std::vector<GradientPair> node_sum_gradients;
  dh::DVec<GradientPair> node_sum_gradients_d;
  /*! \brief row offset in SparsePage (the input data). */
  thrust::device_vector<size_t> row_ptrs;
  /*! \brief On-device feature set, only actually used on one of the devices */
  thrust::device_vector<int> feature_set_d;
  /*! The row offset for this shard. */
  bst_uint row_begin_idx;
  bst_uint row_end_idx;
  bst_uint n_rows;
  int n_bins;

  TrainParam param;
  bool prediction_cache_initialised;

  dh::CubMemory temp_memory;

  std::unique_ptr<GPUHistBuilderBase<GradientSumT>> hist_builder;

  // TODO(canonizer): do add support multi-batch DMatrix here
  DeviceShard(int device_id, bst_uint row_begin, bst_uint row_end,
              TrainParam _param)
      : device_id_(device_id),
        row_begin_idx(row_begin),
        row_end_idx(row_end),
        row_stride(0),
        n_rows(row_end - row_begin),
        n_bins(0),
        null_gidx_value(0),
        param(_param),
        prediction_cache_initialised(false) {}

  /* Init row_ptrs and row_stride */
  void InitRowPtrs(const SparsePage& row_batch) {
    dh::safe_cuda(hipSetDevice(device_id_));
    const auto& offset_vec = row_batch.offset.HostVector();
    row_ptrs.resize(n_rows + 1);
    thrust::copy(offset_vec.data() + row_begin_idx,
                 offset_vec.data() + row_end_idx + 1,
                 row_ptrs.begin());
    auto row_iter = row_ptrs.begin();
    // find the maximum row size for converting to ELLPack
    auto get_size = [=] __device__(size_t row) {
      return row_iter[row + 1] - row_iter[row];
    }; // NOLINT

    auto counting = thrust::make_counting_iterator(size_t(0));
    using TransformT = thrust::transform_iterator<decltype(get_size),
      decltype(counting), size_t>;
    TransformT row_size_iter = TransformT(counting, get_size);
    row_stride = thrust::reduce(row_size_iter, row_size_iter + n_rows, 0,
                                thrust::maximum<size_t>());
  }

  /*
     Init:
     n_bins, null_gidx_value, gidx_buffer, row_ptrs, gidx, gidx_fvalue_map,
     min_fvalue, feature_segments, node_sum_gradients, ridx_segments,
     hist
  */
  void InitCompressedData(
      const common::HistCutMatrix& hmat, const SparsePage& row_batch);

  void CreateHistIndices(const SparsePage& row_batch);

  ~DeviceShard() {
  }

  // Reset values for each update iteration
  void Reset(HostDeviceVector<GradientPair>* dh_gpair) {
    dh::safe_cuda(hipSetDevice(device_id_));
    position.CurrentDVec().Fill(0);
    std::fill(node_sum_gradients.begin(), node_sum_gradients.end(),
              GradientPair());

    thrust::sequence(ridx.CurrentDVec().tbegin(), ridx.CurrentDVec().tend());

    std::fill(ridx_segments.begin(), ridx_segments.end(), Segment(0, 0));
    ridx_segments.front() = Segment(0, ridx.Size());
    this->gpair.copy(dh_gpair->tcbegin(device_id_),
                     dh_gpair->tcend(device_id_));
    SubsampleGradientPair(&gpair, param.subsample, row_begin_idx);
    hist.Reset();
  }

  DeviceSplitCandidate EvaluateSplit(int nidx,
                                     const std::vector<int>& feature_set,
                                     ValueConstraint value_constraint) {
    dh::safe_cuda(hipSetDevice(device_id_));
    auto d_split_candidates = temp_memory.GetSpan<DeviceSplitCandidate>(feature_set.size());
    feature_set_d.resize(feature_set.size());
    auto d_features = common::Span<int>(feature_set_d.data().get(),
                                        feature_set_d.size());
    dh::safe_cuda(hipMemcpy(d_features.data(), feature_set.data(),
                             d_features.size_bytes(), hipMemcpyDefault));
    DeviceNodeStats node(node_sum_gradients[nidx], nidx, param);

    // One block for each feature
    int constexpr BLOCK_THREADS = 256;
    EvaluateSplitKernel<BLOCK_THREADS, GradientSumT>
      <<<uint32_t(feature_set.size()), BLOCK_THREADS, 0>>>
      (hist.GetNodeHistogram(nidx), d_features, node,
       cut_.feature_segments.GetSpan(), cut_.min_fvalue.GetSpan(),
       cut_.gidx_fvalue_map.GetSpan(), GPUTrainingParam(param),
       d_split_candidates, value_constraint, monotone_constraints.GetSpan());

    dh::safe_cuda(hipDeviceSynchronize());
    std::vector<DeviceSplitCandidate> split_candidates(feature_set.size());
    dh::safe_cuda(hipMemcpy(split_candidates.data(), d_split_candidates.data(),
                             split_candidates.size() * sizeof(DeviceSplitCandidate),
                             hipMemcpyDeviceToHost));
    DeviceSplitCandidate best_split;
    for (auto candidate : split_candidates) {
      best_split.Update(candidate, param);
    }
    return best_split;
  }

  void BuildHist(int nidx) {
    hist.AllocateHistogram(nidx);
    hist_builder->Build(this, nidx);
  }

  void SubtractionTrick(int nidx_parent, int nidx_histogram,
                        int nidx_subtraction) {
    auto d_node_hist_parent = hist.GetNodeHistogram(nidx_parent);
    auto d_node_hist_histogram = hist.GetNodeHistogram(nidx_histogram);
    auto d_node_hist_subtraction = hist.GetNodeHistogram(nidx_subtraction);

    dh::LaunchN(device_id_, hist.n_bins, [=] __device__(size_t idx) {
      d_node_hist_subtraction[idx] =
          d_node_hist_parent[idx] - d_node_hist_histogram[idx];
    });
  }

  bool CanDoSubtractionTrick(int nidx_parent, int nidx_histogram,
                             int nidx_subtraction) {
    // Make sure histograms are already allocated
    hist.AllocateHistogram(nidx_subtraction);
    return hist.HistogramExists(nidx_histogram) &&
           hist.HistogramExists(nidx_parent);
  }

  void UpdatePosition(int nidx, int left_nidx, int right_nidx, int fidx,
                      int64_t split_gidx, bool default_dir_left, bool is_dense,
                      int fidx_begin,  // cut.row_ptr[fidx]
                      int fidx_end) {  // cut.row_ptr[fidx + 1]
    dh::safe_cuda(hipSetDevice(device_id_));
    Segment segment = ridx_segments[nidx];
    bst_uint* d_ridx = ridx.Current();
    int* d_position = position.Current();
    common::CompressedIterator<uint32_t> d_gidx = gidx;
    size_t row_stride = this->row_stride;
    // Launch 1 thread for each row
    dh::LaunchN<1, 128>(
        device_id_, segment.Size(), [=] __device__(bst_uint idx) {
          idx += segment.begin;
          bst_uint ridx = d_ridx[idx];
          auto row_begin = row_stride * ridx;
          auto row_end = row_begin + row_stride;
          auto gidx = -1;
          if (is_dense) {
            // FIXME: Maybe just search the cuts again.
            gidx = d_gidx[row_begin + fidx];
          } else {
            gidx = BinarySearchRow(row_begin, row_end, d_gidx, fidx_begin,
                                   fidx_end);
          }

          // belong to left node or right node.
          int position;
          if (gidx >= 0) {
            // Feature is found
            position = gidx <= split_gidx ? left_nidx : right_nidx;
          } else {
            // Feature is missing
            position = default_dir_left ? left_nidx : right_nidx;
          }

          d_position[idx] = position;
        });
    IndicateLeftTransform conversion_op(left_nidx);
    hipcub::TransformInputIterator<int, IndicateLeftTransform, int*> left_itr(
        d_position + segment.begin, conversion_op);
    int left_count = dh::SumReduction(temp_memory, left_itr, segment.Size());
    CHECK_LE(left_count, segment.Size());
    CHECK_GE(left_count, 0);

    SortPositionAndCopy(segment, left_nidx, right_nidx, left_count);

    ridx_segments[left_nidx] =
        Segment(segment.begin, segment.begin + left_count);
    ridx_segments[right_nidx] =
        Segment(segment.begin + left_count, segment.end);
  }

  /*! \brief Sort row indices according to position. */
  void SortPositionAndCopy(const Segment& segment, int left_nidx, int right_nidx,
                       size_t left_count) {
    SortPosition(
        &temp_memory,
        common::Span<int>(position.Current() + segment.begin, segment.Size()),
        common::Span<int>(position.other() + segment.begin, segment.Size()),
        common::Span<bst_uint>(ridx.Current() + segment.begin, segment.Size()),
        common::Span<bst_uint>(ridx.other() + segment.begin, segment.Size()),
        left_nidx, right_nidx, left_count);
    // Copy back key
    dh::safe_cuda(hipMemcpy(
        position.Current() + segment.begin, position.other() + segment.begin,
        segment.Size() * sizeof(int), hipMemcpyDeviceToDevice));
    // Copy back value
    dh::safe_cuda(hipMemcpy(
        ridx.Current() + segment.begin, ridx.other() + segment.begin,
        segment.Size() * sizeof(bst_uint), hipMemcpyDeviceToDevice));
  }

  void UpdatePredictionCache(bst_float* out_preds_d) {
    dh::safe_cuda(hipSetDevice(device_id_));
    if (!prediction_cache_initialised) {
      dh::safe_cuda(hipMemcpy(
          prediction_cache.Data(), out_preds_d,
          prediction_cache.Size() * sizeof(bst_float), hipMemcpyDefault));
    }
    prediction_cache_initialised = true;

    CalcWeightTrainParam param_d(param);

    dh::safe_cuda(hipMemcpy(node_sum_gradients_d.Data(),
                             node_sum_gradients.data(),
                             sizeof(GradientPair) * node_sum_gradients.size(),
                             hipMemcpyHostToDevice));
    auto d_position = position.Current();
    auto d_ridx = ridx.Current();
    auto d_node_sum_gradients = node_sum_gradients_d.Data();
    auto d_prediction_cache = prediction_cache.Data();

    dh::LaunchN(
        device_id_, prediction_cache.Size(), [=] __device__(int local_idx) {
          int pos = d_position[local_idx];
          bst_float weight = CalcWeight(param_d, d_node_sum_gradients[pos]);
          d_prediction_cache[d_ridx[local_idx]] +=
              weight * param_d.learning_rate;
        });

    dh::safe_cuda(hipMemcpy(
        out_preds_d, prediction_cache.Data(),
        prediction_cache.Size() * sizeof(bst_float), hipMemcpyDefault));
  }
};

template <typename GradientSumT>
struct SharedMemHistBuilder : public GPUHistBuilderBase<GradientSumT> {
  void Build(DeviceShard<GradientSumT>* shard, int nidx) override {
    auto segment = shard->ridx_segments[nidx];
    auto segment_begin = segment.begin;
    auto d_node_hist = shard->hist.GetNodeHistogram(nidx);
    auto d_gidx = shard->gidx;
    auto d_ridx = shard->ridx.Current();
    auto d_gpair = shard->gpair.Data();

    int null_gidx_value = shard->null_gidx_value;
    auto n_elements = segment.Size() * shard->row_stride;

    const size_t smem_size = sizeof(GradientSumT) * shard->null_gidx_value;
    const int items_per_thread = 8;
    const int block_threads = 256;
    const int grid_size =
        static_cast<int>(dh::DivRoundUp(n_elements,
                                        items_per_thread * block_threads));
    if (grid_size <= 0) {
      return;
    }
    dh::safe_cuda(hipSetDevice(shard->device_id_));
    SharedMemHistKernel<<<grid_size, block_threads, smem_size>>>
        (shard->row_stride, d_ridx, d_gidx, null_gidx_value, d_node_hist.data(), d_gpair,
         segment_begin, n_elements);
  }
};

template <typename GradientSumT>
struct GlobalMemHistBuilder : public GPUHistBuilderBase<GradientSumT> {
  void Build(DeviceShard<GradientSumT>* shard, int nidx) override {
    Segment segment = shard->ridx_segments[nidx];
    auto d_node_hist = shard->hist.GetNodeHistogram(nidx).data();
    common::CompressedIterator<uint32_t> d_gidx = shard->gidx;
    bst_uint* d_ridx = shard->ridx.Current();
    GradientPair* d_gpair = shard->gpair.Data();

    size_t const n_elements = segment.Size() * shard->row_stride;
    size_t const row_stride = shard->row_stride;
    int const null_gidx_value = shard->null_gidx_value;

    dh::LaunchN(shard->device_id_, n_elements, [=] __device__(size_t idx) {
        int ridx = d_ridx[(idx / row_stride) + segment.begin];
        // lookup the index (bin) of histogram.
        int gidx = d_gidx[ridx * row_stride + idx % row_stride];

        if (gidx != null_gidx_value) {
          AtomicAddGpair(d_node_hist + gidx, d_gpair[ridx]);
        }
      });
  }
};

template <typename GradientSumT>
inline void DeviceShard<GradientSumT>::InitCompressedData(
    const common::HistCutMatrix& hmat, const SparsePage& row_batch) {
  n_bins = hmat.row_ptr.back();
  null_gidx_value = hmat.row_ptr.back();

  int max_nodes =
      param.max_leaves > 0 ? param.max_leaves * 2 : MaxNodesDepth(param.max_depth);

  ba.Allocate(device_id_,
              &gpair, n_rows,
              &ridx, n_rows,
              &position, n_rows,
              &prediction_cache, n_rows,
              &node_sum_gradients_d, max_nodes,
              &cut_.feature_segments, hmat.row_ptr.size(),
              &cut_.gidx_fvalue_map, hmat.cut.size(),
              &cut_.min_fvalue, hmat.min_val.size(),
              &monotone_constraints, param.monotone_constraints.size());
  cut_.gidx_fvalue_map = hmat.cut;
  cut_.min_fvalue = hmat.min_val;
  cut_.feature_segments = hmat.row_ptr;
  monotone_constraints = param.monotone_constraints;

  node_sum_gradients.resize(max_nodes);
  ridx_segments.resize(max_nodes);

  dh::safe_cuda(hipSetDevice(device_id_));

  // allocate compressed bin data
  int num_symbols = n_bins + 1;
  // Required buffer size for storing data matrix in ELLPack format.
  size_t compressed_size_bytes =
      common::CompressedBufferWriter::CalculateBufferSize(row_stride * n_rows,
                                                          num_symbols);

  CHECK(!(param.max_leaves == 0 && param.max_depth == 0))
      << "Max leaves and max depth cannot both be unconstrained for "
      "gpu_hist.";
  ba.Allocate(device_id_, &gidx_buffer, compressed_size_bytes);
  gidx_buffer.Fill(0);

  int nbits = common::detail::SymbolBits(num_symbols);

  CreateHistIndices(row_batch);

  gidx = common::CompressedIterator<uint32_t>(gidx_buffer.Data(), num_symbols);

  // check if we can use shared memory for building histograms
  // (assuming atleast we need 2 CTAs per SM to maintain decent latency hiding)
  auto histogram_size = sizeof(GradientSumT) * null_gidx_value;
  auto max_smem = dh::MaxSharedMemory(device_id_);
  if (histogram_size <= max_smem) {
    hist_builder.reset(new SharedMemHistBuilder<GradientSumT>);
  } else {
    hist_builder.reset(new GlobalMemHistBuilder<GradientSumT>);
  }

  // Init histogram
  hist.Init(device_id_, hmat.row_ptr.back());
}


template <typename GradientSumT>
inline void DeviceShard<GradientSumT>::CreateHistIndices(const SparsePage& row_batch) {
  int num_symbols = n_bins + 1;
  // bin and compress entries in batches of rows
  size_t gpu_batch_nrows =
      std::min
      (dh::TotalMemory(device_id_) / (16 * row_stride * sizeof(Entry)),
       static_cast<size_t>(n_rows));
  const std::vector<Entry>& data_vec = row_batch.data.HostVector();

  thrust::device_vector<Entry> entries_d(gpu_batch_nrows * row_stride);
  size_t gpu_nbatches = dh::DivRoundUp(n_rows, gpu_batch_nrows);

  for (size_t gpu_batch = 0; gpu_batch < gpu_nbatches; ++gpu_batch) {
    size_t batch_row_begin = gpu_batch * gpu_batch_nrows;
    size_t batch_row_end = (gpu_batch + 1) * gpu_batch_nrows;
    if (batch_row_end > n_rows) {
      batch_row_end = n_rows;
    }
    size_t batch_nrows = batch_row_end - batch_row_begin;
    // number of entries in this batch.
    size_t n_entries = row_ptrs[batch_row_end] - row_ptrs[batch_row_begin];
    // copy data entries to device.
    dh::safe_cuda
        (hipMemcpy
         (entries_d.data().get(), data_vec.data() + row_ptrs[batch_row_begin],
          n_entries * sizeof(Entry), hipMemcpyDefault));
    const dim3 block3(32, 8, 1);  // 256 threads
    const dim3 grid3(dh::DivRoundUp(n_rows, block3.x),
                     dh::DivRoundUp(row_stride, block3.y), 1);
    compress_bin_ellpack_k<<<grid3, block3>>>
        (common::CompressedBufferWriter(num_symbols),
         gidx_buffer.Data(),
         row_ptrs.data().get() + batch_row_begin,
         entries_d.data().get(),
         cut_.gidx_fvalue_map.Data(), cut_.feature_segments.Data(),
         batch_row_begin, batch_nrows,
         row_ptrs[batch_row_begin],
         row_stride, null_gidx_value);

    dh::safe_cuda(hipGetLastError());
    dh::safe_cuda(hipDeviceSynchronize());
  }

  // free the memory that is no longer needed
  row_ptrs.resize(0);
  row_ptrs.shrink_to_fit();
  entries_d.resize(0);
  entries_d.shrink_to_fit();
}


template <typename GradientSumT>
class GPUHistMakerSpecialised{
 public:
  struct ExpandEntry;

  GPUHistMakerSpecialised() : initialised_(false), p_last_fmat_(nullptr) {}
  void Init(
      const std::vector<std::pair<std::string, std::string>>& args) {
    param_.InitAllowUnknown(args);
    hist_maker_param_.InitAllowUnknown(args);
    CHECK(param_.n_gpus != 0) << "Must have at least one device";
    n_devices_ = param_.n_gpus;
    dist_ = GPUDistribution::Block(GPUSet::All(param_.gpu_id, param_.n_gpus));

    dh::CheckComputeCapability();

    if (param_.grow_policy == TrainParam::kLossGuide) {
      qexpand_.reset(new ExpandQueue(LossGuide));
    } else {
      qexpand_.reset(new ExpandQueue(DepthWise));
    }

    monitor_.Init("updater_gpu_hist");
  }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) {
    monitor_.Start("Update", dist_.Devices());
    GradStats::CheckInfo(dmat->Info());
    // rescale learning rate according to size of trees
    float lr = param_.learning_rate;
    param_.learning_rate = lr / trees.size();
    ValueConstraint::Init(&param_, dmat->Info().num_col_);
    // build tree
    try {
      for (size_t i = 0; i < trees.size(); ++i) {
        this->UpdateTree(gpair, dmat, trees[i]);
      }
      dh::safe_cuda(hipGetLastError());
    } catch (const std::exception& e) {
      LOG(FATAL) << "Exception in gpu_hist: " << e.what() << std::endl;
    }
    param_.learning_rate = lr;
    monitor_.Stop("Update", dist_.Devices());
  }

  void InitDataOnce(DMatrix* dmat) {
    info_ = &dmat->Info();

    int n_devices = dist_.Devices().Size();

    device_list_.resize(n_devices);
    for (int index = 0; index < n_devices; ++index) {
      int device_id = dist_.Devices().DeviceId(index);
      device_list_[index] = device_id;
    }

    reducer_.Init(device_list_);

    auto batch_iter = dmat->GetRowBatches().begin();
    const SparsePage& batch = *batch_iter;
    // Create device shards
    shards_.resize(n_devices);
    dh::ExecuteIndexShards(&shards_, [&](int i, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
        size_t start = dist_.ShardStart(info_->num_row_, i);
        size_t size = dist_.ShardSize(info_->num_row_, i);
        shard = std::unique_ptr<DeviceShard<GradientSumT>>
                (new DeviceShard<GradientSumT>(dist_.Devices().DeviceId(i),
                                 start, start + size, param_));
        shard->InitRowPtrs(batch);
      });

    // Find the cuts.
    monitor_.Start("Quantiles", dist_.Devices());
    common::DeviceSketch(batch, *info_, param_, &hmat_, hist_maker_param_.gpu_batch_nrows);
    n_bins_ = hmat_.row_ptr.back();
    monitor_.Stop("Quantiles", dist_.Devices());

    monitor_.Start("BinningCompression", dist_.Devices());
    dh::ExecuteIndexShards(&shards_, [&](int idx,
      std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
        shard->InitCompressedData(hmat_, batch);
      });
    monitor_.Stop("BinningCompression", dist_.Devices());
    ++batch_iter;
    CHECK(batch_iter.AtEnd()) << "External memory not supported";

    p_last_fmat_ = dmat;
    initialised_ = true;
  }

  void InitData(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat) {
    monitor_.Start("InitDataOnce", dist_.Devices());
    if (!initialised_) {
      this->InitDataOnce(dmat);
    }
    monitor_.Stop("InitDataOnce", dist_.Devices());

    column_sampler_.Init(info_->num_col_, param_.colsample_bynode,
                         param_.colsample_bylevel, param_.colsample_bytree);

    // Copy gpair & reset memory
    monitor_.Start("InitDataReset", dist_.Devices());

    gpair->Reshard(dist_);
    dh::ExecuteIndexShards(
        &shards_,
        [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
          shard->Reset(gpair);
        });
    monitor_.Stop("InitDataReset", dist_.Devices());
  }

  void AllReduceHist(int nidx) {
    if (shards_.size() == 1) return;

    monitor_.Start("AllReduce");
    dh::ExecuteIndexShards(
        &shards_,
        [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
          auto d_node_hist = shard->hist.GetNodeHistogram(nidx).data();
          reducer_.AllReduceSum(
              dist_.Devices().Index(shard->device_id_),
              reinterpret_cast<typename GradientSumT::ValueT*>(d_node_hist),
              reinterpret_cast<typename GradientSumT::ValueT*>(d_node_hist),
              n_bins_ * (sizeof(GradientSumT) /
                         sizeof(typename GradientSumT::ValueT)));
        });
    monitor_.Stop("AllReduce");
  }

  /**
   * \brief Build GPU local histograms for the left and right child of some parent node
   */
  void BuildHistLeftRight(int nidx_parent, int nidx_left, int nidx_right) {
    size_t left_node_max_elements = 0;
    size_t right_node_max_elements = 0;
    for (auto& shard : shards_) {
      left_node_max_elements = (std::max)(
        left_node_max_elements, shard->ridx_segments[nidx_left].Size());
      right_node_max_elements = (std::max)(
        right_node_max_elements, shard->ridx_segments[nidx_right].Size());
    }

    auto build_hist_nidx = nidx_left;
    auto subtraction_trick_nidx = nidx_right;

    if (right_node_max_elements < left_node_max_elements) {
      build_hist_nidx = nidx_right;
      subtraction_trick_nidx = nidx_left;
    }

    // Build histogram for node with the smallest number of training examples
    dh::ExecuteIndexShards(
        &shards_,
        [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
          shard->BuildHist(build_hist_nidx);
        });

    this->AllReduceHist(build_hist_nidx);

    // Check whether we can use the subtraction trick to calculate the other
    bool do_subtraction_trick = true;
    for (auto& shard : shards_) {
      do_subtraction_trick &= shard->CanDoSubtractionTrick(
        nidx_parent, build_hist_nidx, subtraction_trick_nidx);
    }

    if (do_subtraction_trick) {
      // Calculate other histogram using subtraction trick
      dh::ExecuteIndexShards(
          &shards_,
          [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
            shard->SubtractionTrick(nidx_parent, build_hist_nidx,
                                    subtraction_trick_nidx);
          });
    } else {
      // Calculate other histogram manually
      dh::ExecuteIndexShards(
          &shards_,
          [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
            shard->BuildHist(subtraction_trick_nidx);
          });

      this->AllReduceHist(subtraction_trick_nidx);
    }
  }

  DeviceSplitCandidate EvaluateSplit(int nidx, RegTree* p_tree) {
    return shards_.front()->EvaluateSplit(
        nidx, *column_sampler_.GetFeatureSet(p_tree->GetDepth(nidx)),
        node_value_constraints_[nidx]);
  }

  void InitRoot(RegTree* p_tree) {
    constexpr int root_nidx = 0;
    // Sum gradients
    std::vector<GradientPair> tmp_sums(shards_.size());

    dh::ExecuteIndexShards(
        &shards_,
        [&](int i, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
          dh::safe_cuda(hipSetDevice(shard->device_id_));
          tmp_sums[i] = dh::SumReduction(
              shard->temp_memory, shard->gpair.Data(), shard->gpair.Size());
        });
    GradientPair sum_gradient =
        std::accumulate(tmp_sums.begin(), tmp_sums.end(), GradientPair());

    // Generate root histogram
    dh::ExecuteIndexShards(
        &shards_,
        [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
          shard->BuildHist(root_nidx);
        });

    this->AllReduceHist(root_nidx);

    // Remember root stats
    p_tree->Stat(root_nidx).sum_hess = sum_gradient.GetHess();
    auto weight = CalcWeight(param_, sum_gradient);
    p_tree->Stat(root_nidx).base_weight = weight;
    (*p_tree)[root_nidx].SetLeaf(param_.learning_rate * weight);

    // Store sum gradients
    for (auto& shard : shards_) {
      shard->node_sum_gradients[root_nidx] = sum_gradient;
    }

    // Initialise root constraint
    node_value_constraints_.resize(p_tree->GetNodes().size());

    // Generate first split
    auto split = this->EvaluateSplit(root_nidx, p_tree);
    qexpand_->push(
        ExpandEntry(root_nidx, p_tree->GetDepth(root_nidx), split, 0));
  }

  void UpdatePosition(const ExpandEntry& candidate, RegTree* p_tree) {
    int nidx = candidate.nid;
    int left_nidx = (*p_tree)[nidx].LeftChild();
    int right_nidx = (*p_tree)[nidx].RightChild();

    // convert floating-point split_pt into corresponding bin_id
    // split_cond = -1 indicates that split_pt is less than all known cut points
    int64_t split_gidx = -1;
    int64_t fidx = candidate.split.findex;
    bool default_dir_left = candidate.split.dir == kLeftDir;
    uint32_t fidx_begin = hmat_.row_ptr[fidx];
    uint32_t fidx_end = hmat_.row_ptr[fidx + 1];
    // split_gidx = i where i is the i^th bin containing split value.
    for (auto i = fidx_begin; i < fidx_end; ++i) {
      if (candidate.split.fvalue == hmat_.cut[i]) {
        split_gidx = static_cast<int64_t>(i);
      }
    }
    auto is_dense = info_->num_nonzero_ == info_->num_row_ * info_->num_col_;

    dh::ExecuteIndexShards(
        &shards_,
        [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
          shard->UpdatePosition(nidx, left_nidx, right_nidx, fidx, split_gidx,
                                default_dir_left, is_dense, fidx_begin,
                                fidx_end);
        });
  }

  void ApplySplit(const ExpandEntry& candidate, RegTree* p_tree) {
    // Add new leaves
    RegTree& tree = *p_tree;
    tree.ExpandNode(candidate.nid, candidate.split.findex, candidate.split.fvalue,
                    candidate.split.dir == kLeftDir);
    auto& parent = tree[candidate.nid];
    tree.Stat(candidate.nid).loss_chg = candidate.split.loss_chg;

    // Set up child constraints
    node_value_constraints_.resize(tree.GetNodes().size());
    GradStats left_stats(param_);
    left_stats.Add(candidate.split.left_sum);
    GradStats right_stats(param_);
    right_stats.Add(candidate.split.right_sum);
    node_value_constraints_[candidate.nid].SetChild(
        param_, parent.SplitIndex(), left_stats, right_stats,
        &node_value_constraints_[parent.LeftChild()],
        &node_value_constraints_[parent.RightChild()]);

    // Configure left child
    auto left_weight =
        node_value_constraints_[parent.LeftChild()].CalcWeight(param_, left_stats);
    tree[parent.LeftChild()].SetLeaf(left_weight * param_.learning_rate, 0);
    tree.Stat(parent.LeftChild()).base_weight = left_weight;
    tree.Stat(parent.LeftChild()).sum_hess = candidate.split.left_sum.GetHess();

    // Configure right child
    auto right_weight =
        node_value_constraints_[parent.RightChild()].CalcWeight(param_, right_stats);
    tree[parent.RightChild()].SetLeaf(right_weight * param_.learning_rate, 0);
    tree.Stat(parent.RightChild()).base_weight = right_weight;
    tree.Stat(parent.RightChild()).sum_hess = candidate.split.right_sum.GetHess();

    // Store sum gradients
    for (auto& shard : shards_) {
      shard->node_sum_gradients[parent.LeftChild()] = candidate.split.left_sum;
      shard->node_sum_gradients[parent.RightChild()] = candidate.split.right_sum;
    }
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* p_fmat,
                  RegTree* p_tree) {
    auto& tree = *p_tree;

    monitor_.Start("InitData", dist_.Devices());
    this->InitData(gpair, p_fmat);
    monitor_.Stop("InitData", dist_.Devices());
    monitor_.Start("InitRoot", dist_.Devices());
    this->InitRoot(p_tree);
    monitor_.Stop("InitRoot", dist_.Devices());

    auto timestamp = qexpand_->size();
    auto num_leaves = 1;

    while (!qexpand_->empty()) {
      ExpandEntry candidate = qexpand_->top();
      qexpand_->pop();
      if (!candidate.IsValid(param_, num_leaves)) continue;

      this->ApplySplit(candidate, p_tree);
      monitor_.Start("UpdatePosition", dist_.Devices());
      this->UpdatePosition(candidate, p_tree);
      monitor_.Stop("UpdatePosition", dist_.Devices());
      num_leaves++;

      int left_child_nidx = tree[candidate.nid].LeftChild();
      int right_child_nidx = tree[candidate.nid].RightChild();

      // Only create child entries if needed
      if (ExpandEntry::ChildIsValid(param_, tree.GetDepth(left_child_nidx),
                                    num_leaves)) {
        monitor_.Start("BuildHist", dist_.Devices());
        this->BuildHistLeftRight(candidate.nid, left_child_nidx,
                                 right_child_nidx);
        monitor_.Stop("BuildHist", dist_.Devices());

        monitor_.Start("EvaluateSplits", dist_.Devices());
        auto left_child_split =
            this->EvaluateSplit(left_child_nidx, p_tree);
        auto right_child_split =
            this->EvaluateSplit(right_child_nidx, p_tree);
        qexpand_->push(ExpandEntry(left_child_nidx,
                                   tree.GetDepth(left_child_nidx), left_child_split,
                                   timestamp++));
        qexpand_->push(ExpandEntry(right_child_nidx,
                                   tree.GetDepth(right_child_nidx), right_child_split,
                                   timestamp++));
        monitor_.Stop("EvaluateSplits", dist_.Devices());
      }
    }
  }

  bool UpdatePredictionCache(
      const DMatrix* data, HostDeviceVector<bst_float>* p_out_preds) {
    monitor_.Start("UpdatePredictionCache", dist_.Devices());
    if (shards_.empty() || p_last_fmat_ == nullptr || p_last_fmat_ != data)
      return false;
    p_out_preds->Reshard(dist_.Devices());
    dh::ExecuteIndexShards(
        &shards_,
        [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
          shard->UpdatePredictionCache(
              p_out_preds->DevicePointer(shard->device_id_));
        });
    monitor_.Stop("UpdatePredictionCache", dist_.Devices());
    return true;
  }

  struct ExpandEntry {
    int nid;
    int depth;
    DeviceSplitCandidate split;
    uint64_t timestamp;
    ExpandEntry(int nid, int depth, const DeviceSplitCandidate& split,
                uint64_t timestamp)
        : nid(nid), depth(depth), split(split), timestamp(timestamp) {}
    bool IsValid(const TrainParam& param, int num_leaves) const {
      if (split.loss_chg <= kRtEps) return false;
      if (split.left_sum.GetHess() == 0 || split.right_sum.GetHess() == 0)
        return false;
      if (param.max_depth > 0 && depth == param.max_depth) return false;
      if (param.max_leaves > 0 && num_leaves == param.max_leaves) return false;
      return true;
    }

    static bool ChildIsValid(const TrainParam& param, int depth,
                             int num_leaves) {
      if (param.max_depth > 0 && depth >= param.max_depth) return false;
      if (param.max_leaves > 0 && num_leaves >= param.max_leaves) return false;
      return true;
    }

    friend std::ostream& operator<<(std::ostream& os, const ExpandEntry& e) {
      os << "ExpandEntry: \n";
      os << "nidx: " << e.nid << "\n";
      os << "depth: " << e.depth << "\n";
      os << "loss: " << e.split.loss_chg << "\n";
      os << "left_sum: " << e.split.left_sum << "\n";
      os << "right_sum: " << e.split.right_sum << "\n";
      return os;
    }
  };

  inline static bool DepthWise(ExpandEntry lhs, ExpandEntry rhs) {
    if (lhs.depth == rhs.depth) {
      return lhs.timestamp > rhs.timestamp;  // favor small timestamp
    } else {
      return lhs.depth > rhs.depth;  // favor small depth
    }
  }
  inline static bool LossGuide(ExpandEntry lhs, ExpandEntry rhs) {
    if (lhs.split.loss_chg == rhs.split.loss_chg) {
      return lhs.timestamp > rhs.timestamp;  // favor small timestamp
    } else {
      return lhs.split.loss_chg < rhs.split.loss_chg;  // favor large loss_chg
    }
  }
  TrainParam param_;
  GPUHistMakerTrainParam hist_maker_param_;
  common::HistCutMatrix hmat_;
  common::GHistIndexMatrix gmat_;
  MetaInfo* info_;
  bool initialised_;
  int n_devices_;
  int n_bins_;

  std::vector<std::unique_ptr<DeviceShard<GradientSumT>>> shards_;
  common::ColumnSampler column_sampler_;
  using ExpandQueue = std::priority_queue<ExpandEntry, std::vector<ExpandEntry>,
    std::function<bool(ExpandEntry, ExpandEntry)>>;
  std::unique_ptr<ExpandQueue> qexpand_;
  common::Monitor monitor_;
  dh::AllReducer reducer_;
  std::vector<ValueConstraint> node_value_constraints_;
  /*! List storing device id. */
  std::vector<int> device_list_;

  DMatrix* p_last_fmat_;
  GPUDistribution dist_;
};

class GPUHistMaker : public TreeUpdater {
 public:
  void Init(
      const std::vector<std::pair<std::string, std::string>>& args) override {
    hist_maker_param_.InitAllowUnknown(args);
    float_maker_.reset();
    double_maker_.reset();
    if (hist_maker_param_.single_precision_histogram) {
      float_maker_.reset(new GPUHistMakerSpecialised<GradientPair>());
      float_maker_->Init(args);
    } else {
      double_maker_.reset(new GPUHistMakerSpecialised<GradientPairPrecise>());
      double_maker_->Init(args);
    }
  }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) override {
    if (hist_maker_param_.single_precision_histogram) {
      float_maker_->Update(gpair, dmat, trees);
    } else {
      double_maker_->Update(gpair, dmat, trees);
    }
  }

  bool UpdatePredictionCache(
      const DMatrix* data, HostDeviceVector<bst_float>* p_out_preds) override {
    if (hist_maker_param_.single_precision_histogram) {
      return float_maker_->UpdatePredictionCache(data, p_out_preds);
    } else {
      return double_maker_->UpdatePredictionCache(data, p_out_preds);
    }
  }

 private:
  GPUHistMakerTrainParam hist_maker_param_;
  std::unique_ptr<GPUHistMakerSpecialised<GradientPair>> float_maker_;
  std::unique_ptr<GPUHistMakerSpecialised<GradientPairPrecise>> double_maker_;
};

XGBOOST_REGISTER_TREE_UPDATER(GPUHistMaker, "grow_gpu_hist")
    .describe("Grow tree with GPU.")
    .set_body([]() { return new GPUHistMaker(); });
}  // namespace tree
}  // namespace xgboost
