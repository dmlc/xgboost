#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2020 XGBoost contributors
 */
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <xgboost/tree_updater.h>
#include <algorithm>
#include <cmath>
#include <memory>
#include <limits>
#include <queue>
#include <utility>
#include <vector>

#include "xgboost/host_device_vector.h"
#include "xgboost/parameter.h"
#include "xgboost/span.h"
#include "xgboost/json.h"

#include "../common/io.h"
#include "../common/device_helpers.cuh"
#include "../common/hist_util.h"
#include "../common/timer.h"
#include "../data/ellpack_page.cuh"

#include "param.h"
#include "updater_gpu_common.cuh"
#include "constraints.cuh"
#include "gpu_hist/feature_groups.cuh"
#include "gpu_hist/gradient_based_sampler.cuh"
#include "gpu_hist/row_partitioner.cuh"
#include "gpu_hist/histogram.cuh"
#include "gpu_hist/evaluate_splits.cuh"
#include "gpu_hist/driver.cuh"

namespace xgboost {
namespace tree {
#if !defined(GTEST_TEST)
DMLC_REGISTRY_FILE_TAG(updater_gpu_hist);
#endif  // !defined(GTEST_TEST)

// training parameters specific to this algorithm
struct GPUHistMakerTrainParam
    : public XGBoostParameter<GPUHistMakerTrainParam> {
  bool single_precision_histogram;
  bool deterministic_histogram;
  bool debug_synchronize;
  // declare parameters
  DMLC_DECLARE_PARAMETER(GPUHistMakerTrainParam) {
    DMLC_DECLARE_FIELD(single_precision_histogram).set_default(false).describe(
        "Use single precision to build histograms.");
    DMLC_DECLARE_FIELD(deterministic_histogram).set_default(true).describe(
        "Pre-round the gradient for obtaining deterministic gradient histogram.");
    DMLC_DECLARE_FIELD(debug_synchronize).set_default(false).describe(
        "Check if all distributed tree are identical after tree construction.");
  }
};
#if !defined(GTEST_TEST)
DMLC_REGISTER_PARAMETER(GPUHistMakerTrainParam);
#endif  // !defined(GTEST_TEST)

/**
 * \struct  DeviceHistogram
 *
 * \summary Data storage for node histograms on device. Automatically expands.
 *
 * \tparam GradientSumT      histogram entry type.
 * \tparam kStopGrowingSize  Do not grow beyond this size
 *
 * \author  Rory
 * \date    28/07/2018
 */
template <typename GradientSumT, size_t kStopGrowingSize = 1 << 26>
class DeviceHistogram {
 private:
  /*! \brief Map nidx to starting index of its histogram. */
  std::map<int, size_t> nidx_map_;
  dh::device_vector<typename GradientSumT::ValueT> data_;
  int n_bins_;
  int device_id_;
  static constexpr size_t kNumItemsInGradientSum =
      sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT);
  static_assert(kNumItemsInGradientSum == 2,
                "Number of items in gradient type should be 2.");

 public:
  void Init(int device_id, int n_bins) {
    this->n_bins_ = n_bins;
    this->device_id_ = device_id;
  }

  void Reset() {
    auto d_data = data_.data().get();
      dh::LaunchN(device_id_, data_.size(),
                  [=] __device__(size_t idx) { d_data[idx] = 0.0f; });
    nidx_map_.clear();
  }
  bool HistogramExists(int nidx) const {
    return nidx_map_.find(nidx) != nidx_map_.cend();
  }
  int Bins() const {
    return n_bins_;
  }
  size_t HistogramSize() const {
    return n_bins_ * kNumItemsInGradientSum;
  }

  dh::device_vector<typename GradientSumT::ValueT>& Data() {
    return data_;
  }

  void AllocateHistogram(int nidx) {
    if (HistogramExists(nidx)) return;
    // Number of items currently used in data
    const size_t used_size = nidx_map_.size() * HistogramSize();
    const size_t new_used_size = used_size + HistogramSize();
    if (data_.size() >= kStopGrowingSize) {
      // Recycle histogram memory
      if (new_used_size <= data_.size()) {
        // no need to remove old node, just insert the new one.
        nidx_map_[nidx] = used_size;
        // memset histogram size in bytes
      } else {
        std::pair<int, size_t> old_entry = *nidx_map_.begin();
        nidx_map_.erase(old_entry.first);
        nidx_map_[nidx] = old_entry.second;
      }
      // Zero recycled memory
      auto d_data = data_.data().get() + nidx_map_[nidx];
      dh::LaunchN(device_id_, n_bins_ * 2,
                  [=] __device__(size_t idx) { d_data[idx] = 0.0f; });
    } else {
      // Append new node histogram
      nidx_map_[nidx] = used_size;
      // Check there is enough memory for another histogram node
      if (data_.size() < new_used_size + HistogramSize()) {
        size_t new_required_memory =
            std::max(data_.size() * 2, HistogramSize());
        data_.resize(new_required_memory);
      }
    }

    CHECK_GE(data_.size(), nidx_map_.size() * HistogramSize());
  }

  /**
   * \summary   Return pointer to histogram memory for a given node.
   * \param nidx    Tree node index.
   * \return    hist pointer.
   */
  common::Span<GradientSumT> GetNodeHistogram(int nidx) {
    CHECK(this->HistogramExists(nidx));
    auto ptr = data_.data().get() + nidx_map_[nidx];
    return common::Span<GradientSumT>(
        reinterpret_cast<GradientSumT*>(ptr), n_bins_);
  }
};

struct CalcWeightTrainParam {
  float min_child_weight;
  float reg_alpha;
  float reg_lambda;
  float max_delta_step;
  float learning_rate;
  XGBOOST_DEVICE explicit CalcWeightTrainParam(const TrainParam& p)
      : min_child_weight(p.min_child_weight),
        reg_alpha(p.reg_alpha),
        reg_lambda(p.reg_lambda),
        max_delta_step(p.max_delta_step),
        learning_rate(p.learning_rate) {}
};

// Manage memory for a single GPU
template <typename GradientSumT>
struct GPUHistMakerDevice {
  int device_id;
  EllpackPageImpl* page;
  BatchParam batch_param;

  std::unique_ptr<RowPartitioner> row_partitioner;
  DeviceHistogram<GradientSumT> hist{};

  common::Span<GradientPair> gpair;

  dh::caching_device_vector<int> monotone_constraints;
  dh::caching_device_vector<bst_float> prediction_cache;

  /*! \brief Sum gradient for each node. */
  std::vector<GradientPair> node_sum_gradients;

  TrainParam param;
  bool deterministic_histogram;

  GradientSumT histogram_rounding;

  dh::PinnedMemory pinned;

  std::vector<hipStream_t> streams{};

  common::Monitor monitor;
  std::vector<ValueConstraint> node_value_constraints;
  common::ColumnSampler column_sampler;
  FeatureInteractionConstraintDevice interaction_constraints;

  std::unique_ptr<GradientBasedSampler> sampler;

  std::unique_ptr<FeatureGroups> feature_groups;

  GPUHistMakerDevice(int _device_id,
                     EllpackPageImpl* _page,
                     bst_uint _n_rows,
                     TrainParam _param,
                     uint32_t column_sampler_seed,
                     uint32_t n_features,
                     bool deterministic_histogram,
                     BatchParam _batch_param)
      : device_id(_device_id),
        page(_page),
        param(std::move(_param)),
        column_sampler(column_sampler_seed),
        interaction_constraints(param, n_features),
        deterministic_histogram{deterministic_histogram},
        batch_param(_batch_param) {
    sampler.reset(new GradientBasedSampler(
        page, _n_rows, batch_param, param.subsample, param.sampling_method));
    if (!param.monotone_constraints.empty()) {
      // Copy assigning an empty vector causes an exception in MSVC debug builds
      monotone_constraints = param.monotone_constraints;
    }
    node_sum_gradients.resize(param.MaxNodes());

    // Init histogram
    hist.Init(device_id, page->Cuts().TotalBins());
    monitor.Init(std::string("GPUHistMakerDevice") + std::to_string(device_id));
    feature_groups.reset(new FeatureGroups(
        page->Cuts(), page->is_dense, dh::MaxSharedMemoryOptin(device_id),
        sizeof(GradientSumT)));
  }

  ~GPUHistMakerDevice() {  // NOLINT
    dh::safe_cuda(hipSetDevice(device_id));
    for (auto& stream : streams) {
      dh::safe_cuda(hipStreamDestroy(stream));
    }
  }

  // Get vector of at least n initialised streams
  std::vector<hipStream_t>& GetStreams(int n) {
    if (n > streams.size()) {
      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamDestroy(stream));
      }

      streams.clear();
      streams.resize(n);

      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamCreate(&stream));
      }
    }

    return streams;
  }

  // Reset values for each update iteration
  // Note that the column sampler must be passed by value because it is not
  // thread safe
  void Reset(HostDeviceVector<GradientPair>* dh_gpair, DMatrix* dmat, int64_t num_columns) {
    this->column_sampler.Init(num_columns, param.colsample_bynode,
      param.colsample_bylevel, param.colsample_bytree);
    dh::safe_cuda(hipSetDevice(device_id));
    this->interaction_constraints.Reset();
    std::fill(node_sum_gradients.begin(), node_sum_gradients.end(),
              GradientPair());

    auto sample = sampler->Sample(dh_gpair->DeviceSpan(), dmat);
    page = sample.page;
    gpair = sample.gpair;

    if (deterministic_histogram) {
      histogram_rounding = CreateRoundingFactor<GradientSumT>(this->gpair);
    } else {
      histogram_rounding = GradientSumT{0.0, 0.0};
    }

    row_partitioner.reset();  // Release the device memory first before reallocating
    row_partitioner.reset(new RowPartitioner(device_id,  sample.sample_rows));
    hist.Reset();
  }


  DeviceSplitCandidate EvaluateRootSplit(GradientPair root_sum) {
    int nidx = 0;
    dh::TemporaryArray<DeviceSplitCandidate> splits_out(1);
    GPUTrainingParam gpu_param(param);
    auto sampled_features = column_sampler.GetFeatureSet(0);
    sampled_features->SetDevice(device_id);
    common::Span<bst_feature_t> feature_set =
        interaction_constraints.Query(sampled_features->DeviceSpan(), nidx);
    auto matrix = page->GetDeviceAccessor(device_id);
    EvaluateSplitInputs<GradientSumT> inputs{
        nidx,
        {root_sum.GetGrad(), root_sum.GetHess()},
        gpu_param,
        feature_set,
        matrix.feature_segments,
        matrix.gidx_fvalue_map,
        matrix.min_fvalue,
        hist.GetNodeHistogram(nidx),
        node_value_constraints[nidx],
        dh::ToSpan(monotone_constraints)};
    EvaluateSingleSplit(dh::ToSpan(splits_out), inputs);
    std::vector<DeviceSplitCandidate> result(1);
    dh::safe_cuda(hipMemcpy(result.data(), splits_out.data().get(),
                             sizeof(DeviceSplitCandidate) * splits_out.size(),
                             hipMemcpyDeviceToHost));
    return result.front();
  }

  void EvaluateLeftRightSplits(
      ExpandEntry candidate, int left_nidx, int right_nidx, const RegTree& tree,
      common::Span<ExpandEntry> pinned_candidates_out) {
    dh::TemporaryArray<DeviceSplitCandidate> splits_out(2);
    GPUTrainingParam gpu_param(param);
    auto left_sampled_features =
        column_sampler.GetFeatureSet(tree.GetDepth(left_nidx));
    left_sampled_features->SetDevice(device_id);
    common::Span<bst_feature_t> left_feature_set =
        interaction_constraints.Query(left_sampled_features->DeviceSpan(),
                                      left_nidx);
    auto right_sampled_features =
        column_sampler.GetFeatureSet(tree.GetDepth(right_nidx));
    right_sampled_features->SetDevice(device_id);
    common::Span<bst_feature_t> right_feature_set =
        interaction_constraints.Query(right_sampled_features->DeviceSpan(),
                                      left_nidx);
    auto matrix = page->GetDeviceAccessor(device_id);

    EvaluateSplitInputs<GradientSumT> left{left_nidx,
                                           {candidate.split.left_sum.GetGrad(),
                                            candidate.split.left_sum.GetHess()},
                                           gpu_param,
                                           left_feature_set,
                                           matrix.feature_segments,
                                           matrix.gidx_fvalue_map,
                                           matrix.min_fvalue,
                                           hist.GetNodeHistogram(left_nidx),
                                           node_value_constraints[left_nidx],
                                           dh::ToSpan(monotone_constraints)};
    EvaluateSplitInputs<GradientSumT> right{
        right_nidx,
        {candidate.split.right_sum.GetGrad(),
         candidate.split.right_sum.GetHess()},
        gpu_param,
        right_feature_set,
        matrix.feature_segments,
        matrix.gidx_fvalue_map,
        matrix.min_fvalue,
        hist.GetNodeHistogram(right_nidx),
        node_value_constraints[right_nidx],
        dh::ToSpan(monotone_constraints)};
    auto d_splits_out = dh::ToSpan(splits_out);
    EvaluateSplits(d_splits_out, left, right);
    dh::TemporaryArray<ExpandEntry> entries(2);
    auto d_entries = entries.data().get();
    dh::LaunchN(device_id, 1, [=] __device__(size_t idx) {
      d_entries[0] =
          ExpandEntry(left_nidx, candidate.depth + 1, d_splits_out[0]);
      d_entries[1] =
          ExpandEntry(right_nidx, candidate.depth + 1, d_splits_out[1]);
    });
    dh::safe_cuda(hipMemcpyAsync(
        pinned_candidates_out.data(), entries.data().get(),
        sizeof(ExpandEntry) * entries.size(), hipMemcpyDeviceToHost));
  }

  void BuildHist(int nidx) {
    hist.AllocateHistogram(nidx);
    auto d_node_hist = hist.GetNodeHistogram(nidx);
    auto d_ridx = row_partitioner->GetRows(nidx);
    BuildGradientHistogram(page->GetDeviceAccessor(device_id),
                           feature_groups->DeviceAccessor(device_id), gpair,
                           d_ridx, d_node_hist, histogram_rounding);
  }

  void SubtractionTrick(int nidx_parent, int nidx_histogram,
                        int nidx_subtraction) {
    auto d_node_hist_parent = hist.GetNodeHistogram(nidx_parent);
    auto d_node_hist_histogram = hist.GetNodeHistogram(nidx_histogram);
    auto d_node_hist_subtraction = hist.GetNodeHistogram(nidx_subtraction);

    dh::LaunchN(device_id, page->Cuts().TotalBins(), [=] __device__(size_t idx) {
      d_node_hist_subtraction[idx] =
          d_node_hist_parent[idx] - d_node_hist_histogram[idx];
    });
  }

  bool CanDoSubtractionTrick(int nidx_parent, int nidx_histogram,
                             int nidx_subtraction) {
    // Make sure histograms are already allocated
    hist.AllocateHistogram(nidx_subtraction);
    return hist.HistogramExists(nidx_histogram) &&
           hist.HistogramExists(nidx_parent);
  }

  void UpdatePosition(int nidx, RegTree::Node split_node) {
    auto d_matrix = page->GetDeviceAccessor(device_id);

    row_partitioner->UpdatePosition(
        nidx, split_node.LeftChild(), split_node.RightChild(),
        [=] __device__(bst_uint ridx) {
          // given a row index, returns the node id it belongs to
          bst_float cut_value =
              d_matrix.GetFvalue(ridx, split_node.SplitIndex());
          // Missing value
          int new_position = 0;
          if (isnan(cut_value)) {
            new_position = split_node.DefaultChild();
          } else {
            if (cut_value <= split_node.SplitCond()) {
              new_position = split_node.LeftChild();
            } else {
              new_position = split_node.RightChild();
            }
          }
          return new_position;
        });
  }

  // After tree update is finished, update the position of all training
  // instances to their final leaf. This information is used later to update the
  // prediction cache
  void FinalisePosition(RegTree const* p_tree, DMatrix* p_fmat) {
    dh::TemporaryArray<RegTree::Node> d_nodes(p_tree->GetNodes().size());
    dh::safe_cuda(hipMemcpy(d_nodes.data().get(), p_tree->GetNodes().data(),
                             d_nodes.size() * sizeof(RegTree::Node),
                             hipMemcpyHostToDevice));

    if (row_partitioner->GetRows().size() != p_fmat->Info().num_row_) {
      row_partitioner.reset();  // Release the device memory first before reallocating
      row_partitioner.reset(new RowPartitioner(device_id, p_fmat->Info().num_row_));
    }
    if (page->n_rows == p_fmat->Info().num_row_) {
      FinalisePositionInPage(page, dh::ToSpan(d_nodes));
    } else {
      for (auto& batch : p_fmat->GetBatches<EllpackPage>(batch_param)) {
        FinalisePositionInPage(batch.Impl(), dh::ToSpan(d_nodes));
      }
    }
  }

  void FinalisePositionInPage(EllpackPageImpl* page, const common::Span<RegTree::Node> d_nodes) {
    auto d_matrix = page->GetDeviceAccessor(device_id);
    row_partitioner->FinalisePosition(
        [=] __device__(size_t row_id, int position) {
      if (!d_matrix.IsInRange(row_id)) {
        return RowPartitioner::kIgnoredTreePosition;
      }
      auto node = d_nodes[position];

      while (!node.IsLeaf()) {
        bst_float element = d_matrix.GetFvalue(row_id, node.SplitIndex());
        // Missing value
        if (isnan(element)) {
          position = node.DefaultChild();
        } else {
          if (element <= node.SplitCond()) {
            position = node.LeftChild();
          } else {
            position = node.RightChild();
          }
        }
        node = d_nodes[position];
      }
      return position;
    });
  }

  void UpdatePredictionCache(bst_float* out_preds_d) {
    dh::safe_cuda(hipSetDevice(device_id));
    auto d_ridx = row_partitioner->GetRows();
    if (prediction_cache.size() != d_ridx.size()) {
      prediction_cache.resize(d_ridx.size());
      dh::safe_cuda(hipMemcpyAsync(prediction_cache.data().get(), out_preds_d,
                                    prediction_cache.size() * sizeof(bst_float),
                                    hipMemcpyDefault));
    }

    CalcWeightTrainParam param_d(param);
    dh::TemporaryArray<GradientPair> device_node_sum_gradients(node_sum_gradients.size());

    dh::safe_cuda(
        hipMemcpyAsync(device_node_sum_gradients.data().get(), node_sum_gradients.data(),
                        sizeof(GradientPair) * node_sum_gradients.size(),
                        hipMemcpyHostToDevice));
    auto d_position = row_partitioner->GetPosition();
    auto d_node_sum_gradients = device_node_sum_gradients.data().get();
    auto d_prediction_cache = prediction_cache.data().get();

    dh::LaunchN(
        device_id, prediction_cache.size(), [=] __device__(int local_idx) {
          int pos = d_position[local_idx];
          bst_float weight = CalcWeight(param_d, d_node_sum_gradients[pos]);
          d_prediction_cache[d_ridx[local_idx]] +=
              weight * param_d.learning_rate;
        });

    dh::safe_cuda(hipMemcpy(
        out_preds_d, prediction_cache.data().get(),
        prediction_cache.size() * sizeof(bst_float), hipMemcpyDefault));
    row_partitioner.reset();
  }

  void AllReduceHist(int nidx, dh::AllReducer* reducer) {
    monitor.Start("AllReduce");
    auto d_node_hist = hist.GetNodeHistogram(nidx).data();
    reducer->AllReduceSum(
        reinterpret_cast<typename GradientSumT::ValueT*>(d_node_hist),
        reinterpret_cast<typename GradientSumT::ValueT*>(d_node_hist),
        page->Cuts().TotalBins() * (sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT)));

    monitor.Stop("AllReduce");
  }

  /**
   * \brief Build GPU local histograms for the left and right child of some parent node
   */
  void BuildHistLeftRight(const ExpandEntry &candidate, int nidx_left,
        int nidx_right, dh::AllReducer* reducer) {
    auto build_hist_nidx = nidx_left;
    auto subtraction_trick_nidx = nidx_right;

    // Decide whether to build the left histogram or right histogram
    // Use sum of Hessian as a heuristic to select node with fewest training instances
    bool fewer_right = candidate.split.right_sum.GetHess() < candidate.split.left_sum.GetHess();
    if (fewer_right) {
      std::swap(build_hist_nidx, subtraction_trick_nidx);
    }

    this->BuildHist(build_hist_nidx);
    this->AllReduceHist(build_hist_nidx, reducer);

    // Check whether we can use the subtraction trick to calculate the other
    bool do_subtraction_trick = this->CanDoSubtractionTrick(
        candidate.nid, build_hist_nidx, subtraction_trick_nidx);

    if (do_subtraction_trick) {
      // Calculate other histogram using subtraction trick
      this->SubtractionTrick(candidate.nid, build_hist_nidx,
                             subtraction_trick_nidx);
    } else {
      // Calculate other histogram manually
      this->BuildHist(subtraction_trick_nidx);
      this->AllReduceHist(subtraction_trick_nidx, reducer);
    }
  }

  void ApplySplit(const ExpandEntry& candidate, RegTree* p_tree) {
    RegTree& tree = *p_tree;

    node_value_constraints.resize(tree.GetNodes().size());
    auto parent_sum = candidate.split.left_sum + candidate.split.right_sum;
    auto base_weight = node_value_constraints[candidate.nid].CalcWeight(
        param, parent_sum);
    auto left_weight = node_value_constraints[candidate.nid].CalcWeight(
                           param, candidate.split.left_sum) *
                       param.learning_rate;
    auto right_weight = node_value_constraints[candidate.nid].CalcWeight(
                            param, candidate.split.right_sum) *
                        param.learning_rate;
    tree.ExpandNode(candidate.nid, candidate.split.findex,
                    candidate.split.fvalue, candidate.split.dir == kLeftDir,
                    base_weight, left_weight, right_weight,
                    candidate.split.loss_chg, parent_sum.GetHess(),
                     candidate.split.left_sum.GetHess(), candidate.split.right_sum.GetHess());
    // Set up child constraints
    node_value_constraints.resize(tree.GetNodes().size());
    node_value_constraints[candidate.nid].SetChild(
        param, tree[candidate.nid].SplitIndex(), candidate.split.left_sum,
        candidate.split.right_sum,
        &node_value_constraints[tree[candidate.nid].LeftChild()],
        &node_value_constraints[tree[candidate.nid].RightChild()]);
    node_sum_gradients[tree[candidate.nid].LeftChild()] =
        candidate.split.left_sum;
    node_sum_gradients[tree[candidate.nid].RightChild()] =
        candidate.split.right_sum;

    interaction_constraints.Split(
        candidate.nid, tree[candidate.nid].SplitIndex(),
        tree[candidate.nid].LeftChild(),
                                  tree[candidate.nid].RightChild());
  }

  ExpandEntry InitRoot(RegTree* p_tree, dh::AllReducer* reducer) {
    constexpr bst_node_t kRootNIdx = 0;
    dh::XGBCachingDeviceAllocator<char> alloc;
    GradientPair root_sum = thrust::reduce(
        thrust::cuda::par(alloc),
        thrust::device_ptr<GradientPair const>(gpair.data()),
        thrust::device_ptr<GradientPair const>(gpair.data() + gpair.size()));
    rabit::Allreduce<rabit::op::Sum, float>(reinterpret_cast<float*>(&root_sum),
                                            2);

    this->BuildHist(kRootNIdx);
    this->AllReduceHist(kRootNIdx, reducer);

    // Remember root stats
    node_sum_gradients[kRootNIdx] = root_sum;
    p_tree->Stat(kRootNIdx).sum_hess = root_sum.GetHess();
    auto weight = CalcWeight(param, root_sum);
    p_tree->Stat(kRootNIdx).base_weight = weight;
    (*p_tree)[kRootNIdx].SetLeaf(param.learning_rate * weight);

    // Initialise root constraint
    node_value_constraints.resize(p_tree->GetNodes().size());

    // Generate first split
    auto split = this->EvaluateRootSplit(root_sum);
    return ExpandEntry(kRootNIdx, p_tree->GetDepth(kRootNIdx), split);
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair_all, DMatrix* p_fmat,
                  RegTree* p_tree, dh::AllReducer* reducer) {
    auto& tree = *p_tree;
    Driver driver(static_cast<TrainParam::TreeGrowPolicy>(param.grow_policy));

    monitor.Start("Reset");
    this->Reset(gpair_all, p_fmat, p_fmat->Info().num_col_);
    monitor.Stop("Reset");

    monitor.Start("InitRoot");
    driver.Push({ this->InitRoot(p_tree, reducer) });
    monitor.Stop("InitRoot");

    auto num_leaves = 1;

    // The set of leaves that can be expanded asynchronously
    auto expand_set = driver.Pop();
    while (!expand_set.empty()) {
      auto new_candidates =
          pinned.GetSpan<ExpandEntry>(expand_set.size() * 2, ExpandEntry());

      for (auto i = 0ull; i < expand_set.size(); i++) {
        auto candidate = expand_set.at(i);
        if (!candidate.IsValid(param, num_leaves)) {
          continue;
        }
        this->ApplySplit(candidate, p_tree);

        num_leaves++;

        int left_child_nidx = tree[candidate.nid].LeftChild();
        int right_child_nidx = tree[candidate.nid].RightChild();
        // Only create child entries if needed
        if (ExpandEntry::ChildIsValid(param, tree.GetDepth(left_child_nidx),
          num_leaves)) {
          monitor.Start("UpdatePosition");
          this->UpdatePosition(candidate.nid, (*p_tree)[candidate.nid]);
          monitor.Stop("UpdatePosition");

          monitor.Start("BuildHist");
          this->BuildHistLeftRight(candidate, left_child_nidx, right_child_nidx, reducer);
          monitor.Stop("BuildHist");

          monitor.Start("EvaluateSplits");
          this->EvaluateLeftRightSplits(candidate, left_child_nidx,
                                        right_child_nidx, *p_tree,
                                        new_candidates.subspan(i * 2, 2));
          monitor.Stop("EvaluateSplits");
        } else {
          // Set default
          new_candidates[i * 2] = ExpandEntry();
          new_candidates[i * 2 + 1] = ExpandEntry();
        }
      }
      dh::safe_cuda(hipDeviceSynchronize());
      driver.Push(new_candidates.begin(), new_candidates.end());
      expand_set = driver.Pop();
    }

    monitor.Start("FinalisePosition");
    this->FinalisePosition(p_tree, p_fmat);
    monitor.Stop("FinalisePosition");
  }
};

template <typename GradientSumT>
class GPUHistMakerSpecialised {
 public:
  GPUHistMakerSpecialised() = default;
  void Configure(const Args& args, GenericParameter const* generic_param) {
    param_.UpdateAllowUnknown(args);
    generic_param_ = generic_param;
    hist_maker_param_.UpdateAllowUnknown(args);
    dh::CheckComputeCapability();

    monitor_.Init("updater_gpu_hist");
  }

  ~GPUHistMakerSpecialised() {  // NOLINT
    dh::GlobalMemoryLogger().Log();
  }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) {
    monitor_.Start("Update");

    // rescale learning rate according to size of trees
    float lr = param_.learning_rate;
    param_.learning_rate = lr / trees.size();
    ValueConstraint::Init(&param_, dmat->Info().num_col_);
    // build tree
    try {
      for (xgboost::RegTree* tree : trees) {
        this->UpdateTree(gpair, dmat, tree);

        if (hist_maker_param_.debug_synchronize) {
          this->CheckTreesSynchronized(tree);
        }
      }
      dh::safe_cuda(hipGetLastError());
    } catch (const std::exception& e) {
      LOG(FATAL) << "Exception in gpu_hist: " << e.what() << std::endl;
    }

    param_.learning_rate = lr;
    monitor_.Stop("Update");
  }

  void InitDataOnce(DMatrix* dmat) {
    device_ = generic_param_->gpu_id;
    CHECK_GE(device_, 0) << "Must have at least one device";
    info_ = &dmat->Info();
    reducer_.Init({device_});  // NOLINT

    // Synchronise the column sampling seed
    uint32_t column_sampling_seed = common::GlobalRandom()();
    rabit::Broadcast(&column_sampling_seed, sizeof(column_sampling_seed), 0);

    BatchParam batch_param{
      device_,
      param_.max_bin,
      generic_param_->gpu_page_size
    };
    auto page = (*dmat->GetBatches<EllpackPage>(batch_param).begin()).Impl();
    dh::safe_cuda(hipSetDevice(device_));
    maker.reset(new GPUHistMakerDevice<GradientSumT>(device_,
                                                     page,
                                                     info_->num_row_,
                                                     param_,
                                                     column_sampling_seed,
                                                     info_->num_col_,
                                                     hist_maker_param_.deterministic_histogram,
                                                     batch_param));

    p_last_fmat_ = dmat;
    initialised_ = true;
  }

  void InitData(DMatrix* dmat) {
    if (!initialised_) {
      monitor_.Start("InitDataOnce");
      this->InitDataOnce(dmat);
      monitor_.Stop("InitDataOnce");
    }
  }

  // Only call this method for testing
  void CheckTreesSynchronized(RegTree* local_tree) const {
    std::string s_model;
    common::MemoryBufferStream fs(&s_model);
    int rank = rabit::GetRank();
    if (rank == 0) {
      local_tree->Save(&fs);
    }
    fs.Seek(0);
    rabit::Broadcast(&s_model, 0);
    RegTree reference_tree {};  // rank 0 tree
    reference_tree.Load(&fs);
    CHECK(*local_tree == reference_tree);
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* p_fmat,
                  RegTree* p_tree) {
    monitor_.Start("InitData");
    this->InitData(p_fmat);
    monitor_.Stop("InitData");

    gpair->SetDevice(device_);
    maker->UpdateTree(gpair, p_fmat, p_tree, &reducer_);
  }

  bool UpdatePredictionCache(const DMatrix* data, HostDeviceVector<bst_float>* p_out_preds) {
    if (maker == nullptr || p_last_fmat_ == nullptr || p_last_fmat_ != data) {
      return false;
    }
    monitor_.Start("UpdatePredictionCache");
    p_out_preds->SetDevice(device_);
    maker->UpdatePredictionCache(p_out_preds->DevicePointer());
    monitor_.Stop("UpdatePredictionCache");
    return true;
  }

  TrainParam param_;   // NOLINT
  MetaInfo* info_{};   // NOLINT

  std::unique_ptr<GPUHistMakerDevice<GradientSumT>> maker;  // NOLINT

 private:
  bool initialised_ { false };

  GPUHistMakerTrainParam hist_maker_param_;
  GenericParameter const* generic_param_;

  dh::AllReducer reducer_;

  DMatrix* p_last_fmat_ { nullptr };
  int device_{-1};

  common::Monitor monitor_;
};

class GPUHistMaker : public TreeUpdater {
 public:
  void Configure(const Args& args) override {
    // Used in test to count how many configurations are performed
    LOG(DEBUG) << "[GPU Hist]: Configure";
    hist_maker_param_.UpdateAllowUnknown(args);
    // The passed in args can be empty, if we simply purge the old maker without
    // preserving parameters then we can't do Update on it.
    TrainParam param;
    if (float_maker_) {
      param = float_maker_->param_;
    } else if (double_maker_) {
      param = double_maker_->param_;
    }
    if (hist_maker_param_.single_precision_histogram) {
      float_maker_.reset(new GPUHistMakerSpecialised<GradientPair>());
      float_maker_->param_ = param;
      float_maker_->Configure(args, tparam_);
    } else {
      double_maker_.reset(new GPUHistMakerSpecialised<GradientPairPrecise>());
      double_maker_->param_ = param;
      double_maker_->Configure(args, tparam_);
    }
  }

  void LoadConfig(Json const& in) override {
    auto const& config = get<Object const>(in);
    FromJson(config.at("gpu_hist_train_param"), &this->hist_maker_param_);
    if (hist_maker_param_.single_precision_histogram) {
      float_maker_.reset(new GPUHistMakerSpecialised<GradientPair>());
      FromJson(config.at("train_param"), &float_maker_->param_);
    } else {
      double_maker_.reset(new GPUHistMakerSpecialised<GradientPairPrecise>());
      FromJson(config.at("train_param"), &double_maker_->param_);
    }
  }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["gpu_hist_train_param"] = ToJson(hist_maker_param_);
    if (hist_maker_param_.single_precision_histogram) {
      out["train_param"] = ToJson(float_maker_->param_);
    } else {
      out["train_param"] = ToJson(double_maker_->param_);
    }
  }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) override {
    if (hist_maker_param_.single_precision_histogram) {
      float_maker_->Update(gpair, dmat, trees);
    } else {
      double_maker_->Update(gpair, dmat, trees);
    }
  }

  bool UpdatePredictionCache(
      const DMatrix* data, HostDeviceVector<bst_float>* p_out_preds) override {
    if (hist_maker_param_.single_precision_histogram) {
      return float_maker_->UpdatePredictionCache(data, p_out_preds);
    } else {
      return double_maker_->UpdatePredictionCache(data, p_out_preds);
    }
  }

  char const* Name() const override {
    return "grow_gpu_hist";
  }

 private:
  GPUHistMakerTrainParam hist_maker_param_;
  std::unique_ptr<GPUHistMakerSpecialised<GradientPair>> float_maker_;
  std::unique_ptr<GPUHistMakerSpecialised<GradientPairPrecise>> double_maker_;
};

#if !defined(GTEST_TEST)
XGBOOST_REGISTER_TREE_UPDATER(GPUHistMaker, "grow_gpu_hist")
    .describe("Grow tree with GPU.")
    .set_body([]() { return new GPUHistMaker(); });
#endif  // !defined(GTEST_TEST)

}  // namespace tree
}  // namespace xgboost
