#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2022 XGBoost contributors
 */
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <xgboost/tree_updater.h>
#include <algorithm>
#include <cmath>
#include <memory>
#include <limits>
#include <utility>
#include <vector>

#include "xgboost/base.h"
#include "xgboost/data.h"
#include "xgboost/generic_parameters.h"
#include "xgboost/host_device_vector.h"
#include "xgboost/parameter.h"
#include "xgboost/span.h"
#include "xgboost/json.h"

#include "../collective/device_communicator.cuh"
#include "../common/io.h"
#include "../common/device_helpers.cuh"
#include "../common/hist_util.h"
#include "../common/bitfield.h"
#include "../common/timer.h"
#include "../common/categorical.h"
#include "../data/ellpack_page.cuh"

#include "param.h"
#include "driver.h"
#include "updater_gpu_common.cuh"
#include "split_evaluator.h"
#include "constraints.cuh"
#include "gpu_hist/feature_groups.cuh"
#include "gpu_hist/gradient_based_sampler.cuh"
#include "gpu_hist/row_partitioner.cuh"
#include "gpu_hist/histogram.cuh"
#include "gpu_hist/evaluate_splits.cuh"
#include "gpu_hist/expand_entry.cuh"
#include "xgboost/task.h"
#include "xgboost/tree_model.h"

namespace xgboost {
namespace tree {
#if !defined(GTEST_TEST)
DMLC_REGISTRY_FILE_TAG(updater_gpu_hist);
#endif  // !defined(GTEST_TEST)

// training parameters specific to this algorithm
struct GPUHistMakerTrainParam
    : public XGBoostParameter<GPUHistMakerTrainParam> {
  bool debug_synchronize;
  // declare parameters
  DMLC_DECLARE_PARAMETER(GPUHistMakerTrainParam) {
    DMLC_DECLARE_FIELD(debug_synchronize).set_default(false).describe(
        "Check if all distributed tree are identical after tree construction.");
  }
};
#if !defined(GTEST_TEST)
DMLC_REGISTER_PARAMETER(GPUHistMakerTrainParam);
#endif  // !defined(GTEST_TEST)

/**
 * \struct  DeviceHistogramStorage
 *
 * \summary Data storage for node histograms on device. Automatically expands.
 *
 * \tparam GradientSumT      histogram entry type.
 * \tparam kStopGrowingSize  Do not grow beyond this size
 *
 * \author  Rory
 * \date    28/07/2018
 */
template <size_t kStopGrowingSize = 1 << 28>
class DeviceHistogramStorage {
 private:
  using GradientSumT = GradientPairInt64;
  /*! \brief Map nidx to starting index of its histogram. */
  std::map<int, size_t> nidx_map_;
  // Large buffer of zeroed memory, caches histograms
  dh::device_vector<typename GradientSumT::ValueT> data_;
  // If we run out of storage allocate one histogram at a time
  // in overflow. Not cached, overwritten when a new histogram
  // is requested
  dh::device_vector<typename GradientSumT::ValueT> overflow_;
  std::map<int, size_t> overflow_nidx_map_;
  int n_bins_;
  int device_id_;
  static constexpr size_t kNumItemsInGradientSum =
      sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT);
  static_assert(kNumItemsInGradientSum == 2, "Number of items in gradient type should be 2.");

 public:
  // Start with about 16mb
  DeviceHistogramStorage() { data_.reserve(1 << 22); }
  void Init(int device_id, int n_bins) {
    this->n_bins_ = n_bins;
    this->device_id_ = device_id;
  }

  void Reset() {
    auto d_data = data_.data().get();
    dh::LaunchN(data_.size(), [=] __device__(size_t idx) { d_data[idx] = 0.0f; });
    nidx_map_.clear();
    overflow_nidx_map_.clear();
  }
  bool HistogramExists(int nidx) const {
    return nidx_map_.find(nidx) != nidx_map_.cend() ||
           overflow_nidx_map_.find(nidx) != overflow_nidx_map_.cend();
  }
  int Bins() const { return n_bins_; }
  size_t HistogramSize() const { return n_bins_ * kNumItemsInGradientSum; }
  dh::device_vector<typename GradientSumT::ValueT>& Data() { return data_; }

  void AllocateHistograms(const std::vector<int>& new_nidxs) {
    for (int nidx : new_nidxs) {
      CHECK(!HistogramExists(nidx));
    }
    // Number of items currently used in data
    const size_t used_size = nidx_map_.size() * HistogramSize();
    const size_t new_used_size = used_size + HistogramSize() * new_nidxs.size();
    if (used_size >= kStopGrowingSize) {
      // Use overflow
      // Delete previous entries
      overflow_nidx_map_.clear();
      overflow_.resize(HistogramSize() * new_nidxs.size());
      // Zero memory
      auto d_data = overflow_.data().get();
      dh::LaunchN(overflow_.size(),
                  [=] __device__(size_t idx) { d_data[idx] = 0.0; });
      // Append new histograms
      for (int nidx : new_nidxs) {
        overflow_nidx_map_[nidx] = overflow_nidx_map_.size() * HistogramSize();
      }
    } else {
      CHECK_GE(data_.size(), used_size);
      // Expand if necessary
      if (data_.size() < new_used_size) {
        data_.resize(std::max(data_.size() * 2, new_used_size));
      }
      // Append new histograms
      for (int nidx : new_nidxs) {
        nidx_map_[nidx] = nidx_map_.size() * HistogramSize();
      }
    }

    CHECK_GE(data_.size(), nidx_map_.size() * HistogramSize());
  }

  /**
   * \summary   Return pointer to histogram memory for a given node.
   * \param nidx    Tree node index.
   * \return    hist pointer.
   */
  common::Span<GradientSumT> GetNodeHistogram(int nidx) {
    CHECK(this->HistogramExists(nidx));

    if (nidx_map_.find(nidx) != nidx_map_.cend()) {
      // Fetch from normal cache
      auto ptr = data_.data().get() + nidx_map_.at(nidx);
      return common::Span<GradientSumT>(reinterpret_cast<GradientSumT*>(ptr), n_bins_);
    } else {
      // Fetch from overflow
      auto ptr = overflow_.data().get() + overflow_nidx_map_.at(nidx);
      return common::Span<GradientSumT>(reinterpret_cast<GradientSumT*>(ptr), n_bins_);
    }
  }
};

// Manage memory for a single GPU
template <typename GradientSumT>
struct GPUHistMakerDevice {
 private:
  GPUHistEvaluator evaluator_;
  Context const* ctx_;

 public:
  EllpackPageImpl const* page;
  common::Span<FeatureType const> feature_types;
  BatchParam batch_param;

  std::unique_ptr<RowPartitioner> row_partitioner;
  DeviceHistogramStorage<> hist{};

  dh::device_vector<GradientPair> d_gpair;  // storage for gpair;
  common::Span<GradientPair> gpair;

  dh::device_vector<int> monotone_constraints;
  dh::device_vector<float> update_predictions;

  TrainParam param;

  std::unique_ptr<GradientQuantiser> quantiser;

  dh::PinnedMemory pinned;
  dh::PinnedMemory pinned2;

  common::Monitor monitor;
  common::ColumnSampler column_sampler;
  FeatureInteractionConstraintDevice interaction_constraints;

  std::unique_ptr<GradientBasedSampler> sampler;

  std::unique_ptr<FeatureGroups> feature_groups;


  GPUHistMakerDevice(Context const* ctx, EllpackPageImpl const* _page,
                     common::Span<FeatureType const> _feature_types, bst_uint _n_rows,
                     TrainParam _param, uint32_t column_sampler_seed, uint32_t n_features,
                     BatchParam _batch_param)
      : evaluator_{_param, n_features, ctx->gpu_id},
        ctx_(ctx),
        page(_page),
        feature_types{_feature_types},
        param(std::move(_param)),
        column_sampler(column_sampler_seed),
        interaction_constraints(param, n_features),
        batch_param(std::move(_batch_param)) {
    sampler.reset(new GradientBasedSampler(page, _n_rows, batch_param, param.subsample,
                                           param.sampling_method));
    if (!param.monotone_constraints.empty()) {
      // Copy assigning an empty vector causes an exception in MSVC debug builds
      monotone_constraints = param.monotone_constraints;
    }

    // Init histogram
    hist.Init(ctx_->gpu_id, page->Cuts().TotalBins());
    monitor.Init(std::string("GPUHistMakerDevice") + std::to_string(ctx_->gpu_id));
    feature_groups.reset(new FeatureGroups(page->Cuts(), page->is_dense,
                                           dh::MaxSharedMemoryOptin(ctx_->gpu_id),
                                           sizeof(GradientSumT)));
  }

  ~GPUHistMakerDevice() {  // NOLINT
    dh::safe_cuda(hipSetDevice(ctx_->gpu_id));
  }

  // Reset values for each update iteration
  // Note that the column sampler must be passed by value because it is not
  // thread safe
  void Reset(HostDeviceVector<GradientPair>* dh_gpair, DMatrix* dmat, int64_t num_columns) {
    auto const& info = dmat->Info();
    this->column_sampler.Init(num_columns, info.feature_weights.HostVector(),
                              param.colsample_bynode, param.colsample_bylevel,
                              param.colsample_bytree);
    dh::safe_cuda(hipSetDevice(ctx_->gpu_id));

    this->evaluator_.Reset(page->Cuts(), feature_types, dmat->Info().num_col_, param,
                           ctx_->gpu_id);

    this->interaction_constraints.Reset();

    if (d_gpair.size() != dh_gpair->Size()) {
      d_gpair.resize(dh_gpair->Size());
    }
    dh::safe_cuda(hipMemcpyAsync(
        d_gpair.data().get(), dh_gpair->ConstDevicePointer(),
        dh_gpair->Size() * sizeof(GradientPair), hipMemcpyDeviceToDevice));
    auto sample = sampler->Sample(dh::ToSpan(d_gpair), dmat);
    page = sample.page;
    gpair = sample.gpair;

    quantiser.reset(new GradientQuantiser(this->gpair));

    row_partitioner.reset();  // Release the device memory first before reallocating
    row_partitioner.reset(new RowPartitioner(ctx_->gpu_id,  sample.sample_rows));
    hist.Reset();
  }

  GPUExpandEntry EvaluateRootSplit(GradientPairInt64 root_sum) {
    int nidx = RegTree::kRoot;
    GPUTrainingParam gpu_param(param);
    auto sampled_features = column_sampler.GetFeatureSet(0);
    sampled_features->SetDevice(ctx_->gpu_id);
    common::Span<bst_feature_t> feature_set =
        interaction_constraints.Query(sampled_features->DeviceSpan(), nidx);
    auto matrix = page->GetDeviceAccessor(ctx_->gpu_id);
    EvaluateSplitInputs inputs{nidx, 0, root_sum, feature_set, hist.GetNodeHistogram(nidx)};
    EvaluateSplitSharedInputs shared_inputs{
        gpu_param,
        *quantiser,
        feature_types,
        matrix.feature_segments,
        matrix.gidx_fvalue_map,
        matrix.min_fvalue,
        matrix.is_dense
    };
    auto split = this->evaluator_.EvaluateSingleSplit(inputs, shared_inputs);
    return split;
  }

  void EvaluateSplits(const std::vector<GPUExpandEntry>& candidates, const RegTree& tree,
                               common::Span<GPUExpandEntry> pinned_candidates_out) {
    if (candidates.empty()) return;
    dh::TemporaryArray<EvaluateSplitInputs> d_node_inputs(2 * candidates.size());
    dh::TemporaryArray<DeviceSplitCandidate> splits_out(2 * candidates.size());
    std::vector<bst_node_t> nidx(2 * candidates.size());
    auto h_node_inputs = pinned2.GetSpan<EvaluateSplitInputs>(2 * candidates.size());
    auto matrix = page->GetDeviceAccessor(ctx_->gpu_id);
    EvaluateSplitSharedInputs shared_inputs{
        GPUTrainingParam{param}, *quantiser, feature_types,     matrix.feature_segments,
        matrix.gidx_fvalue_map,  matrix.min_fvalue,
        matrix.is_dense
    };
    dh::TemporaryArray<GPUExpandEntry> entries(2 * candidates.size());
    for (size_t i = 0; i < candidates.size(); i++) {
      auto candidate = candidates.at(i);
      int left_nidx = tree[candidate.nid].LeftChild();
      int right_nidx = tree[candidate.nid].RightChild();
      nidx[i * 2] = left_nidx;
      nidx[i * 2 + 1] = right_nidx;
      auto left_sampled_features = column_sampler.GetFeatureSet(tree.GetDepth(left_nidx));
      left_sampled_features->SetDevice(ctx_->gpu_id);
      common::Span<bst_feature_t> left_feature_set =
          interaction_constraints.Query(left_sampled_features->DeviceSpan(), left_nidx);
      auto right_sampled_features = column_sampler.GetFeatureSet(tree.GetDepth(right_nidx));
      right_sampled_features->SetDevice(ctx_->gpu_id);
      common::Span<bst_feature_t> right_feature_set =
          interaction_constraints.Query(right_sampled_features->DeviceSpan(), left_nidx);
      h_node_inputs[i * 2] = {left_nidx, candidate.depth + 1, candidate.split.left_sum,
                              left_feature_set, hist.GetNodeHistogram(left_nidx)};
      h_node_inputs[i * 2 + 1] = {right_nidx, candidate.depth + 1, candidate.split.right_sum,
                                  right_feature_set, hist.GetNodeHistogram(right_nidx)};
    }
    bst_feature_t number_active_features = h_node_inputs[0].feature_set.size();
    for (auto input : h_node_inputs) {
      CHECK_EQ(input.feature_set.size(), number_active_features)
          << "Current implementation assumes that the number of active features "
             "(after sampling) in any node is the same";
    }
    dh::safe_cuda(hipMemcpyAsync(d_node_inputs.data().get(), h_node_inputs.data(),
                                  h_node_inputs.size() * sizeof(EvaluateSplitInputs),
                                  hipMemcpyDefault));

    this->evaluator_.EvaluateSplits(nidx, number_active_features, dh::ToSpan(d_node_inputs),
                                    shared_inputs, dh::ToSpan(entries));
    dh::safe_cuda(hipMemcpyAsync(pinned_candidates_out.data(),
                                  entries.data().get(), sizeof(GPUExpandEntry) * entries.size(),
                                  hipMemcpyDeviceToHost));
    dh::DefaultStream().Sync();
    }

  void BuildHist(int nidx) {
    auto d_node_hist = hist.GetNodeHistogram(nidx);
    auto d_ridx = row_partitioner->GetRows(nidx);
    BuildGradientHistogram(page->GetDeviceAccessor(ctx_->gpu_id),
                           feature_groups->DeviceAccessor(ctx_->gpu_id), gpair,
                           d_ridx, d_node_hist, *quantiser);
  }

  // Attempt to do subtraction trick
  // return true if succeeded
  bool SubtractionTrick(int nidx_parent, int nidx_histogram, int nidx_subtraction) {
    if (!hist.HistogramExists(nidx_histogram) || !hist.HistogramExists(nidx_parent)) {
      return false;
    }
    auto d_node_hist_parent = hist.GetNodeHistogram(nidx_parent);
    auto d_node_hist_histogram = hist.GetNodeHistogram(nidx_histogram);
    auto d_node_hist_subtraction = hist.GetNodeHistogram(nidx_subtraction);

    dh::LaunchN(page->Cuts().TotalBins(), [=] __device__(size_t idx) {
      d_node_hist_subtraction[idx] =
          d_node_hist_parent[idx] - d_node_hist_histogram[idx];
    });
    return true;
  }

  // Extra data for each node that is passed
  // to the update position function
  struct NodeSplitData {
    RegTree::Node split_node;
    FeatureType split_type;
    common::CatBitField node_cats;
  };

  void UpdatePosition(const std::vector<GPUExpandEntry>& candidates, RegTree* p_tree) {
    if (candidates.empty()) return;
    std::vector<int> nidx(candidates.size());
    std::vector<int> left_nidx(candidates.size());
    std::vector<int> right_nidx(candidates.size());
    std::vector<NodeSplitData> split_data(candidates.size());
    for (size_t i = 0; i < candidates.size(); i++) {
      auto& e = candidates[i];
      RegTree::Node split_node = (*p_tree)[e.nid];
      auto split_type = p_tree->NodeSplitType(e.nid);
      nidx.at(i) = e.nid;
      left_nidx.at(i) = split_node.LeftChild();
      right_nidx.at(i) = split_node.RightChild();
      split_data.at(i) = NodeSplitData{split_node, split_type, e.split.split_cats};
    }

    auto d_matrix = page->GetDeviceAccessor(ctx_->gpu_id);
    row_partitioner->UpdatePositionBatch(
        nidx, left_nidx, right_nidx, split_data,
        [=] __device__(bst_uint ridx, const NodeSplitData& data) {
          // given a row index, returns the node id it belongs to
          bst_float cut_value = d_matrix.GetFvalue(ridx, data.split_node.SplitIndex());
          // Missing value
          bool go_left = true;
          if (isnan(cut_value)) {
            go_left = data.split_node.DefaultLeft();
          } else {
            if (data.split_type == FeatureType::kCategorical) {
              go_left = common::Decision<false>(data.node_cats.Bits(), cut_value,
                                                data.split_node.DefaultLeft());
            } else {
              go_left = cut_value <= data.split_node.SplitCond();
            }
          }
          return go_left;
        });
  }

  // After tree update is finished, update the position of all training
  // instances to their final leaf. This information is used later to update the
  // prediction cache
  void FinalisePosition(RegTree const* p_tree, DMatrix* p_fmat, ObjInfo task,
                        HostDeviceVector<bst_node_t>* p_out_position) {
    // Prediction cache will not be used with external memory
    if (!p_fmat->SingleColBlock()) {
      if (task.UpdateTreeLeaf()) {
        LOG(FATAL) << "Current objective function can not be used with external memory.";
      }
      p_out_position->Resize(0);
      update_predictions.clear();
      return;
    }

    dh::TemporaryArray<RegTree::Node> d_nodes(p_tree->GetNodes().size());
    dh::safe_cuda(hipMemcpyAsync(d_nodes.data().get(), p_tree->GetNodes().data(),
                                  d_nodes.size() * sizeof(RegTree::Node),
                                  hipMemcpyHostToDevice));
    auto const& h_split_types = p_tree->GetSplitTypes();
    auto const& categories = p_tree->GetSplitCategories();
    auto const& categories_segments = p_tree->GetSplitCategoriesPtr();

    dh::caching_device_vector<FeatureType> d_split_types;
    dh::caching_device_vector<uint32_t> d_categories;
    dh::caching_device_vector<RegTree::Segment> d_categories_segments;

    if (!categories.empty()) {
      dh::CopyToD(h_split_types, &d_split_types);
      dh::CopyToD(categories, &d_categories);
      dh::CopyToD(categories_segments, &d_categories_segments);
    }

    FinalisePositionInPage(page, dh::ToSpan(d_nodes), dh::ToSpan(d_split_types),
                           dh::ToSpan(d_categories), dh::ToSpan(d_categories_segments),
                           p_out_position);
  }

  void FinalisePositionInPage(EllpackPageImpl const *page,
                              const common::Span<RegTree::Node> d_nodes,
                              common::Span<FeatureType const> d_feature_types,
                              common::Span<uint32_t const> categories,
                              common::Span<RegTree::Segment> categories_segments,
                              HostDeviceVector<bst_node_t>* p_out_position) {
    auto d_matrix = page->GetDeviceAccessor(ctx_->gpu_id);
    auto d_gpair = this->gpair;
    update_predictions.resize(row_partitioner->GetRows().size());
    auto d_update_predictions = dh::ToSpan(update_predictions);
    p_out_position->SetDevice(ctx_->gpu_id);
    p_out_position->Resize(row_partitioner->GetRows().size());

    auto new_position_op = [=] __device__(size_t row_id, int position) {
      // What happens if user prune the tree?
      if (!d_matrix.IsInRange(row_id)) {
        return RowPartitioner::kIgnoredTreePosition;
      }
      auto node = d_nodes[position];

      while (!node.IsLeaf()) {
        bst_float element = d_matrix.GetFvalue(row_id, node.SplitIndex());
        // Missing value
        if (isnan(element)) {
          position = node.DefaultChild();
        } else {
          bool go_left = true;
          if (common::IsCat(d_feature_types, position)) {
            auto node_cats = categories.subspan(categories_segments[position].beg,
                                                categories_segments[position].size);
            go_left = common::Decision<false>(node_cats, element, node.DefaultLeft());
          } else {
            go_left = element <= node.SplitCond();
          }
          if (go_left) {
            position = node.LeftChild();
          } else {
            position = node.RightChild();
          }
        }

        node = d_nodes[position];
      }

      d_update_predictions[row_id] = node.LeafValue();
      return position;
    };  // NOLINT

    auto d_out_position = p_out_position->DeviceSpan();
    row_partitioner->FinalisePosition(d_out_position, new_position_op);

    dh::LaunchN(row_partitioner->GetRows().size(), [=] __device__(size_t idx) {
      bst_node_t position = d_out_position[idx];
      d_update_predictions[idx] = d_nodes[position].LeafValue();
      bool is_row_sampled = d_gpair[idx].GetHess() - .0f == 0.f;
      d_out_position[idx] = is_row_sampled ? ~position : position;
    });
  }

  bool UpdatePredictionCache(linalg::VectorView<float> out_preds_d, RegTree const* p_tree) {
    if (update_predictions.empty()) {
      return false;
    }
    CHECK(p_tree);
    dh::safe_cuda(hipSetDevice(ctx_->gpu_id));
    CHECK_EQ(out_preds_d.DeviceIdx(), ctx_->gpu_id);
    auto d_update_predictions = dh::ToSpan(update_predictions);
    CHECK_EQ(out_preds_d.Size(), d_update_predictions.size());
    dh::LaunchN(out_preds_d.Size(), [=] XGBOOST_DEVICE(size_t idx) mutable {
      out_preds_d(idx) += d_update_predictions[idx];
    });
    return true;
  }

  // num histograms is the number of contiguous histograms in memory to reduce over
  void AllReduceHist(int nidx, collective::DeviceCommunicator* communicator, int num_histograms) {
    monitor.Start("AllReduce");
    auto d_node_hist = hist.GetNodeHistogram(nidx).data();
    using ReduceT = typename std::remove_pointer<decltype(d_node_hist)>::type::ValueT;
    communicator->AllReduceSum(reinterpret_cast<ReduceT*>(d_node_hist),
                               page->Cuts().TotalBins() * 2 * num_histograms);

    monitor.Stop("AllReduce");
  }

  /**
   * \brief Build GPU local histograms for the left and right child of some parent node
   */
  void BuildHistLeftRight(std::vector<GPUExpandEntry> const& candidates,
                          collective::DeviceCommunicator* communicator, const RegTree& tree) {
    if (candidates.empty()) return;
    // Some nodes we will manually compute histograms
    // others we will do by subtraction
    std::vector<int> hist_nidx;
    std::vector<int> subtraction_nidx;
    for (auto& e : candidates) {
      // Decide whether to build the left histogram or right histogram
      // Use sum of Hessian as a heuristic to select node with fewest training instances
      bool fewer_right = e.split.right_sum.GetQuantisedHess() < e.split.left_sum.GetQuantisedHess();
      if (fewer_right) {
        hist_nidx.emplace_back(tree[e.nid].RightChild());
        subtraction_nidx.emplace_back(tree[e.nid].LeftChild());
      } else {
        hist_nidx.emplace_back(tree[e.nid].LeftChild());
        subtraction_nidx.emplace_back(tree[e.nid].RightChild());
      }
    }
    std::vector<int> all_new = hist_nidx;
    all_new.insert(all_new.end(), subtraction_nidx.begin(), subtraction_nidx.end());
    // Allocate the histograms
    // Guaranteed contiguous memory
    hist.AllocateHistograms(all_new);

    for (auto nidx : hist_nidx) {
      this->BuildHist(nidx);
    }

    // Reduce all in one go
    // This gives much better latency in a distributed setting
    // when processing a large batch
    this->AllReduceHist(hist_nidx.at(0), communicator, hist_nidx.size());

    for (size_t i = 0; i < subtraction_nidx.size(); i++) {
      auto build_hist_nidx = hist_nidx.at(i);
      auto subtraction_trick_nidx = subtraction_nidx.at(i);
      auto parent_nidx = candidates.at(i).nid;

      if (!this->SubtractionTrick(parent_nidx, build_hist_nidx, subtraction_trick_nidx)) {
        // Calculate other histogram manually
        this->BuildHist(subtraction_trick_nidx);
        this->AllReduceHist(subtraction_trick_nidx, communicator, 1);
      }
    }
  }

  void ApplySplit(const GPUExpandEntry& candidate, RegTree* p_tree) {
    RegTree& tree = *p_tree;

    // Sanity check - have we created a leaf with no training instances?
    if (!collective::IsDistributed() && row_partitioner) {
      CHECK(row_partitioner->GetRows(candidate.nid).size() > 0)
          << "No training instances in this leaf!";
    }

    auto base_weight = candidate.base_weight;
    auto left_weight = candidate.left_weight * param.learning_rate;
    auto right_weight = candidate.right_weight * param.learning_rate;
    auto parent_hess = quantiser
                           ->ToFloatingPoint(candidate.split.left_sum +
                                             candidate.split.right_sum)
                           .GetHess();
    auto left_hess =
        quantiser->ToFloatingPoint(candidate.split.left_sum).GetHess();
    auto right_hess =
        quantiser->ToFloatingPoint(candidate.split.right_sum).GetHess();

    auto is_cat = candidate.split.is_cat;
    if (is_cat) {
      // should be set to nan in evaluation split.
      CHECK(common::CheckNAN(candidate.split.fvalue));
      std::vector<common::CatBitField::value_type> split_cats;

      CHECK_GT(candidate.split.split_cats.Bits().size(), 0);
      auto h_cats = this->evaluator_.GetHostNodeCats(candidate.nid);
      auto n_bins_feature = page->Cuts().FeatureBins(candidate.split.findex);
      split_cats.resize(common::CatBitField::ComputeStorageSize(n_bins_feature), 0);
      CHECK_LE(split_cats.size(), h_cats.size());
      std::copy(h_cats.data(), h_cats.data() + split_cats.size(), split_cats.data());

      tree.ExpandCategorical(
          candidate.nid, candidate.split.findex, split_cats, candidate.split.dir == kLeftDir,
          base_weight, left_weight, right_weight, candidate.split.loss_chg, parent_hess,
          left_hess, right_hess);
    } else {
      CHECK(!common::CheckNAN(candidate.split.fvalue));
      tree.ExpandNode(candidate.nid, candidate.split.findex, candidate.split.fvalue,
                      candidate.split.dir == kLeftDir, base_weight, left_weight, right_weight,
                      candidate.split.loss_chg, parent_hess,
          left_hess, right_hess);
    }
    evaluator_.ApplyTreeSplit(candidate, p_tree);

    const auto& parent = tree[candidate.nid];
    std::size_t max_nidx = std::max(parent.LeftChild(), parent.RightChild());
    interaction_constraints.Split(candidate.nid, parent.SplitIndex(), parent.LeftChild(),
                                  parent.RightChild());
  }

  GPUExpandEntry InitRoot(RegTree* p_tree, collective::DeviceCommunicator* communicator) {
    constexpr bst_node_t kRootNIdx = 0;
    dh::XGBCachingDeviceAllocator<char> alloc;
    auto quantiser = *this->quantiser;
    auto gpair_it = dh::MakeTransformIterator<GradientPairInt64>(
        dh::tbegin(gpair), [=] __device__(auto const &gpair) {
          return quantiser.ToFixedPoint(gpair);
        });
    GradientPairInt64 root_sum_quantised =
        dh::Reduce(thrust::cuda::par(alloc), gpair_it, gpair_it + gpair.size(),
                   GradientPairInt64{}, thrust::plus<GradientPairInt64>{});
    using ReduceT = typename decltype(root_sum_quantised)::ValueT;
    collective::Allreduce<collective::Operation::kSum>(
        reinterpret_cast<ReduceT *>(&root_sum_quantised), 2);

    hist.AllocateHistograms({kRootNIdx});
    this->BuildHist(kRootNIdx);
    this->AllReduceHist(kRootNIdx, communicator, 1);

    // Remember root stats
    auto root_sum = quantiser.ToFloatingPoint(root_sum_quantised);
    p_tree->Stat(kRootNIdx).sum_hess = root_sum.GetHess();
    auto weight = CalcWeight(param, root_sum);
    p_tree->Stat(kRootNIdx).base_weight = weight;
    (*p_tree)[kRootNIdx].SetLeaf(param.learning_rate * weight);

    // Generate first split
    auto root_entry = this->EvaluateRootSplit(root_sum_quantised);
    return root_entry;
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair_all, DMatrix* p_fmat, ObjInfo task,
                  RegTree* p_tree, collective::DeviceCommunicator* communicator,
                  HostDeviceVector<bst_node_t>* p_out_position) {
    auto& tree = *p_tree;
    // Process maximum 32 nodes at a time
    Driver<GPUExpandEntry> driver(param, 32);

    monitor.Start("Reset");
    this->Reset(gpair_all, p_fmat, p_fmat->Info().num_col_);
    monitor.Stop("Reset");

    monitor.Start("InitRoot");
    driver.Push({ this->InitRoot(p_tree, communicator) });
    monitor.Stop("InitRoot");

    // The set of leaves that can be expanded asynchronously
    auto expand_set = driver.Pop();
    while (!expand_set.empty()) {
      for (auto& candidate : expand_set) {
        this->ApplySplit(candidate, p_tree);
      }
      // Get the candidates we are allowed to expand further
      // e.g. We do not bother further processing nodes whose children are beyond max depth
      std::vector<GPUExpandEntry> filtered_expand_set;
      std::copy_if(expand_set.begin(), expand_set.end(), std::back_inserter(filtered_expand_set),
                   [&](const auto& e) { return driver.IsChildValid(e); });


      auto new_candidates =
          pinned.GetSpan<GPUExpandEntry>(filtered_expand_set.size() * 2, GPUExpandEntry());

      monitor.Start("UpdatePosition");
      // Update position is only run when child is valid, instead of right after apply
      // split (as in approx tree method).  Hense we have the finalise position call
      // in GPU Hist.
      this->UpdatePosition(filtered_expand_set, p_tree);
      monitor.Stop("UpdatePosition");

      monitor.Start("BuildHist");
      this->BuildHistLeftRight(filtered_expand_set, communicator, tree);
      monitor.Stop("BuildHist");

      monitor.Start("EvaluateSplits");
      this->EvaluateSplits(filtered_expand_set, *p_tree, new_candidates);
      monitor.Stop("EvaluateSplits");
      dh::DefaultStream().Sync();
      driver.Push(new_candidates.begin(), new_candidates.end());
      expand_set = driver.Pop();
    }

    monitor.Start("FinalisePosition");
    this->FinalisePosition(p_tree, p_fmat, task, p_out_position);
    monitor.Stop("FinalisePosition");
  }
};

class GPUHistMaker : public TreeUpdater {
  using GradientSumT = GradientPairPrecise;

 public:
  explicit GPUHistMaker(GenericParameter const* ctx, ObjInfo task)
      : TreeUpdater(ctx), task_{task} {};
  void Configure(const Args& args) override {
    // Used in test to count how many configurations are performed
    LOG(DEBUG) << "[GPU Hist]: Configure";
    param_.UpdateAllowUnknown(args);
    hist_maker_param_.UpdateAllowUnknown(args);
    dh::CheckComputeCapability();
    initialised_ = false;

    monitor_.Init("updater_gpu_hist");
  }

  void LoadConfig(Json const& in) override {
    auto const& config = get<Object const>(in);
    FromJson(config.at("gpu_hist_train_param"), &this->hist_maker_param_);
    initialised_ = false;
    FromJson(config.at("train_param"), &param_);
  }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["gpu_hist_train_param"] = ToJson(hist_maker_param_);
    out["train_param"] = ToJson(param_);
  }

  ~GPUHistMaker() {  // NOLINT
    dh::GlobalMemoryLogger().Log();
  }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              common::Span<HostDeviceVector<bst_node_t>> out_position,
              const std::vector<RegTree*>& trees) override {
    monitor_.Start("Update");

    // rescale learning rate according to size of trees
    float lr = param_.learning_rate;
    param_.learning_rate = lr / trees.size();

    // build tree
    try {
      size_t t_idx{0};
      for (xgboost::RegTree* tree : trees) {
        this->UpdateTree(gpair, dmat, tree, &out_position[t_idx]);

        if (hist_maker_param_.debug_synchronize) {
          this->CheckTreesSynchronized(tree);
        }
        ++t_idx;
      }
      dh::safe_cuda(hipGetLastError());
    } catch (const std::exception& e) {
      LOG(FATAL) << "Exception in gpu_hist: " << e.what() << std::endl;
    }

    param_.learning_rate = lr;
    monitor_.Stop("Update");
  }

  void InitDataOnce(DMatrix* dmat) {
    CHECK_GE(ctx_->gpu_id, 0) << "Must have at least one device";
    info_ = &dmat->Info();

    // Synchronise the column sampling seed
    uint32_t column_sampling_seed = common::GlobalRandom()();
    collective::Broadcast(&column_sampling_seed, sizeof(column_sampling_seed), 0);

    BatchParam batch_param{
      ctx_->gpu_id,
      param_.max_bin,
    };
    auto page = (*dmat->GetBatches<EllpackPage>(batch_param).begin()).Impl();
    dh::safe_cuda(hipSetDevice(ctx_->gpu_id));
    info_->feature_types.SetDevice(ctx_->gpu_id);
    maker.reset(new GPUHistMakerDevice<GradientSumT>(
        ctx_, page, info_->feature_types.ConstDeviceSpan(), info_->num_row_, param_,
        column_sampling_seed, info_->num_col_, batch_param));

    p_last_fmat_ = dmat;
    initialised_ = true;
  }

  void InitData(DMatrix* dmat, RegTree const* p_tree) {
    if (!initialised_) {
      monitor_.Start("InitDataOnce");
      this->InitDataOnce(dmat);
      monitor_.Stop("InitDataOnce");
    }
    p_last_tree_ = p_tree;
  }

  // Only call this method for testing
  void CheckTreesSynchronized(RegTree* local_tree) const {
    std::string s_model;
    common::MemoryBufferStream fs(&s_model);
    int rank = collective::GetRank();
    if (rank == 0) {
      local_tree->Save(&fs);
    }
    fs.Seek(0);
    collective::Broadcast(&s_model, 0);
    RegTree reference_tree{};  // rank 0 tree
    reference_tree.Load(&fs);
    CHECK(*local_tree == reference_tree);
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* p_fmat, RegTree* p_tree,
                  HostDeviceVector<bst_node_t>* p_out_position) {
    monitor_.Start("InitData");
    this->InitData(p_fmat, p_tree);
    monitor_.Stop("InitData");

    gpair->SetDevice(ctx_->gpu_id);
    auto* communicator = collective::Communicator::GetDevice(ctx_->gpu_id);
    maker->UpdateTree(gpair, p_fmat, task_, p_tree, communicator, p_out_position);
  }

  bool UpdatePredictionCache(const DMatrix* data,
                             linalg::VectorView<bst_float> p_out_preds) override {
    if (maker == nullptr || p_last_fmat_ == nullptr || p_last_fmat_ != data) {
      return false;
    }
    monitor_.Start("UpdatePredictionCache");
    bool result = maker->UpdatePredictionCache(p_out_preds, p_last_tree_);
    monitor_.Stop("UpdatePredictionCache");
    return result;
  }

  TrainParam param_;  // NOLINT
  MetaInfo* info_{};  // NOLINT

  std::unique_ptr<GPUHistMakerDevice<GradientSumT>> maker;  // NOLINT

  char const* Name() const override { return "grow_gpu_hist"; }

 private:
  bool initialised_{false};

  GPUHistMakerTrainParam hist_maker_param_;

  DMatrix* p_last_fmat_{nullptr};
  RegTree const* p_last_tree_{nullptr};
  ObjInfo task_;

  common::Monitor monitor_;
};

#if !defined(GTEST_TEST)
XGBOOST_REGISTER_TREE_UPDATER(GPUHistMaker, "grow_gpu_hist")
    .describe("Grow tree with GPU.")
    .set_body([](GenericParameter const* tparam, ObjInfo task) {
      return new GPUHistMaker(tparam, task);
    });
#endif  // !defined(GTEST_TEST)

}  // namespace tree
}  // namespace xgboost
