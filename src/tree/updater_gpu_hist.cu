#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2019 XGBoost contributors
 */
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <xgboost/tree_updater.h>
#include <algorithm>
#include <cmath>
#include <memory>
#include <limits>
#include <queue>
#include <utility>
#include <vector>
#include "../common/common.h"
#include "../common/compressed_iterator.h"
#include "../common/device_helpers.cuh"
#include "../common/hist_util.h"
#include "../common/host_device_vector.h"
#include "../common/timer.h"
#include "../common/span.h"
#include "param.h"
#include "updater_gpu_common.cuh"
#include "constraints.cuh"
#include "gpu_hist/row_partitioner.cuh"

namespace xgboost {
namespace tree {

#if !defined(GTEST_TEST)
DMLC_REGISTRY_FILE_TAG(updater_gpu_hist);
#endif  // !defined(GTEST_TEST)

// training parameters specific to this algorithm
struct GPUHistMakerTrainParam
    : public dmlc::Parameter<GPUHistMakerTrainParam> {
  bool single_precision_histogram;
  // number of rows in a single GPU batch
  int gpu_batch_nrows;
  bool debug_synchronize;
  // declare parameters
  DMLC_DECLARE_PARAMETER(GPUHistMakerTrainParam) {
    DMLC_DECLARE_FIELD(single_precision_histogram).set_default(false).describe(
        "Use single precision to build histograms.");
    DMLC_DECLARE_FIELD(gpu_batch_nrows)
        .set_lower_bound(-1)
        .set_default(0)
        .describe("Number of rows in a GPU batch, used for finding quantiles on GPU; "
                  "-1 to use all rows assignted to a GPU, and 0 to auto-deduce");
    DMLC_DECLARE_FIELD(debug_synchronize).set_default(false).describe(
        "Check if all distributed tree are identical after tree construction.");
  }
};
#if !defined(GTEST_TEST)
DMLC_REGISTER_PARAMETER(GPUHistMakerTrainParam);
#endif  // !defined(GTEST_TEST)

struct ExpandEntry {
  int nid;
  int depth;
  DeviceSplitCandidate split;
  uint64_t timestamp;
  ExpandEntry() = default;
  ExpandEntry(int nid, int depth, DeviceSplitCandidate split,
              uint64_t timestamp)
      : nid(nid), depth(depth), split(std::move(split)), timestamp(timestamp) {}
  bool IsValid(const TrainParam& param, int num_leaves) const {
    if (split.loss_chg <= kRtEps) return false;
    if (split.left_sum.GetHess() == 0 || split.right_sum.GetHess() == 0) {
      return false;
    }
    if (param.max_depth > 0 && depth == param.max_depth) return false;
    if (param.max_leaves > 0 && num_leaves == param.max_leaves) return false;
    return true;
  }

  static bool ChildIsValid(const TrainParam& param, int depth, int num_leaves) {
    if (param.max_depth > 0 && depth >= param.max_depth) return false;
    if (param.max_leaves > 0 && num_leaves >= param.max_leaves) return false;
    return true;
  }

  friend std::ostream& operator<<(std::ostream& os, const ExpandEntry& e) {
    os << "ExpandEntry: \n";
    os << "nidx: " << e.nid << "\n";
    os << "depth: " << e.depth << "\n";
    os << "loss: " << e.split.loss_chg << "\n";
    os << "left_sum: " << e.split.left_sum << "\n";
    os << "right_sum: " << e.split.right_sum << "\n";
    return os;
  }
};

inline static bool DepthWise(ExpandEntry lhs, ExpandEntry rhs) {
  if (lhs.depth == rhs.depth) {
    return lhs.timestamp > rhs.timestamp;  // favor small timestamp
  } else {
    return lhs.depth > rhs.depth;  // favor small depth
  }
}
inline static bool LossGuide(ExpandEntry lhs, ExpandEntry rhs) {
  if (lhs.split.loss_chg == rhs.split.loss_chg) {
    return lhs.timestamp > rhs.timestamp;  // favor small timestamp
  } else {
    return lhs.split.loss_chg < rhs.split.loss_chg;  // favor large loss_chg
  }
}

// Find a gidx value for a given feature otherwise return -1 if not found
__forceinline__ __device__ int BinarySearchRow(
    bst_uint begin, bst_uint end,
    common::CompressedIterator<uint32_t> data,
    int const fidx_begin, int const fidx_end) {
  bst_uint previous_middle = UINT32_MAX;
  while (end != begin) {
    auto middle = begin + (end - begin) / 2;
    if (middle == previous_middle) {
      break;
    }
    previous_middle = middle;

    auto gidx = data[middle];

    if (gidx >= fidx_begin && gidx < fidx_end) {
      return gidx;
    } else if (gidx < fidx_begin) {
      begin = middle;
    } else {
      end = middle;
    }
  }
  // Value is missing
  return -1;
}

// A context that is created for every row that is processed during binning. This is then
// handed off to the different matrices to write to the underlying stream it manages
struct CompressRowContext {
  int bin_;  // NOLINT: Feature bin
  size_t irow_;  // NOLINT: Row to process
  size_t base_row_;  // NOLINT: Total number of rows processed thus far
  size_t row_offset_in_batch_;  // NOLINT: Offset to current row in the batch
  int ifeature_;  // NOLINT: Feature to process
  size_t base_item_offset_;  // NOLINT: Offset to the item in the current batch
  size_t total_items_processed_;  // NOLINT: Total number of items processed thus far

  __device__ explicit CompressRowContext(
    int bin, size_t irow, size_t base_row, size_t row_offset_in_batch,
    int ifeature, size_t base_item_offset, size_t total_items_processed)
    : bin_(bin), irow_(irow), base_row_(base_row), row_offset_in_batch_(row_offset_in_batch),
      ifeature_(ifeature), base_item_offset_(base_item_offset),
      total_items_processed_(total_items_processed) {}
};

/*! \brief How is the compressed data laid out? */
enum class CompressedDataLayout {
  kRowStride,  // Every row is evenly sized with row stride number of items
  kCSR  // Every row is sized based on the actual number of items in that row
};

// Base type of all matrices containing the histograms for all the features that are needed
// for binning. It also abstracts some of the feature quantile properties
struct MatrixBase {
  common::Span<uint32_t> feature_segments;
  /*! \brief minimum value for each feature. */
  common::Span<bst_float> min_fvalue;
  /*! \brief Cut. */
  common::Span<bst_float> gidx_fvalue_map;
  int null_gidx_value;

  /*! \brief row length for ELLPack. */
  size_t row_stride{0};

  common::CompressedBufferWriter gidx_buffer_writer;
  common::CompressedIterator<uint32_t> gidx_buffer_iter;
  common::Span<common::CompressedByteT> gidx_buffer;

  __device__  explicit MatrixBase(
    common::Span<uint32_t> fsegs, common::Span<bst_float> min_fvals,
    common::Span<bst_float> fval_map, common::CompressedBufferWriter buf_wr,
    common::CompressedIterator<uint32_t> buf_itr, common::Span<common::CompressedByteT> buf,
    int ngidx, size_t rstride)
      : feature_segments(fsegs), min_fvalue(min_fvals), gidx_fvalue_map(fval_map),
        null_gidx_value(ngidx), gidx_buffer_writer(buf_wr), gidx_buffer_iter(buf_itr),
        gidx_buffer(buf), row_stride(rstride) {}
  __device__  virtual ~MatrixBase() {}  // NOLINT

  __forceinline__ __device__ virtual bst_float GetElement(size_t ridx, size_t fidx) const = 0;
  __forceinline__ __device__ virtual int GetGidx(size_t ridx, size_t gidx_pos) const {
    return gidx_buffer_iter[ridx * row_stride + gidx_pos % row_stride];
  }
  __forceinline__ __device__ virtual void Write(const CompressRowContext &com_ctx) {
    gidx_buffer_writer.AtomicWriteSymbol(
      gidx_buffer.data(), com_ctx.bin_,
      (com_ctx.irow_ + com_ctx.base_row_) * row_stride + com_ctx.ifeature_);
  }

  __forceinline__ __device__ uint32_t GetFeatureBin(int fidx) const {
    return feature_segments[fidx];
  }
  __forceinline__ __device__ bst_float GetMinFeatureValue(int fidx) const {
    return min_fvalue[fidx];
  }
  __forceinline__ __device__ const bst_float *GetFeatureValue(int fbin) const {
    return &gidx_fvalue_map[fbin];
  }
  __forceinline__ __device__ size_t BinCount() const { return gidx_fvalue_map.size(); }
  __forceinline__ __device__ size_t RowStride() const { return row_stride; }
  __forceinline__ __device__ int NullGidxValue() const { return null_gidx_value; }
};

// A dense matrix representation, where every row contains every feature
struct DenseMatrix : MatrixBase {
  __forceinline__ __device__ bst_float GetElement(size_t ridx, size_t fidx) const override {
    auto row_begin = row_stride * ridx;
    auto gidx = gidx_buffer_iter[row_begin + fidx];
    return gidx_fvalue_map[gidx];
  }

  __device__  explicit DenseMatrix(
    common::Span<uint32_t> fsegs, common::Span<bst_float> min_fvals,
    common::Span<bst_float> fval_map, common::CompressedBufferWriter buf_wr,
    common::CompressedIterator<uint32_t> buf_itr, common::Span<common::CompressedByteT> buf,
    int ngidx, size_t rstride)
      : MatrixBase(fsegs, min_fvals, fval_map, buf_wr, buf_itr, buf, ngidx, rstride) {}
};

// A sparse matrix representation, where each row contains a constant number of features
struct RowStrideMatrix : MatrixBase {
  __forceinline__ __device__ bst_float GetElement(size_t ridx, size_t fidx) const override {
    auto row_begin = row_stride * ridx;
    auto row_end = row_begin + row_stride;
    auto gidx = BinarySearchRow(row_begin, row_end, gidx_buffer_iter, feature_segments[fidx],
                                feature_segments[fidx + 1]);
    return (gidx == -1) ? nan("") : gidx_fvalue_map[gidx];
  }

  __device__  explicit RowStrideMatrix(
    common::Span<uint32_t> fsegs, common::Span<bst_float> min_fvals,
    common::Span<bst_float> fval_map, common::CompressedBufferWriter buf_wr,
    common::CompressedIterator<uint32_t> buf_itr, common::Span<common::CompressedByteT> buf,
    int ngidx, size_t rstride)
      : MatrixBase(fsegs, min_fvals, fval_map, buf_wr, buf_itr, buf, ngidx, rstride) {}
};

// A sparse matrix representation in the CSR format, where it contains the exact number of items
// present in the matrix. A sparse matrix can either be a RowStrideMatrix/CSRMatrix based on
// which representation consumes less GPU memory
struct CSRMatrix : MatrixBase {
  common::CompressedBufferWriter gidx_row_writer;
  common::CompressedIterator<uint32_t> gidx_row_iter;
  common::Span<common::CompressedByteT> gidx_row_buffer;

  size_t n_rows;  // Number of rows in this matrix
  size_t n_items;  // Number of items in this matrix

  __forceinline__ __device__ bst_float GetElement(size_t ridx, size_t fidx) const override {
    auto row_begin = gidx_row_iter[ridx];
    auto row_end = gidx_row_iter[ridx + 1];
    auto gidx = BinarySearchRow(row_begin, row_end, gidx_buffer_iter, feature_segments[fidx],
                                feature_segments[fidx + 1]);
    return (gidx == -1) ? nan("") : gidx_fvalue_map[gidx];
  }

  __forceinline__ __device__ int GetGidx(size_t ridx, size_t gidx_pos) const override {
    uint32_t n_elems = gidx_row_iter[ridx + 1] - gidx_row_iter[ridx];
    if (gidx_pos % row_stride < n_elems) {
      return gidx_buffer_iter[gidx_row_iter[ridx] + gidx_pos % row_stride];
    }
    return null_gidx_value;
  }

  __forceinline__ __device__ void Write(const CompressRowContext &com_ctx) override {
    if (com_ctx.bin_ != null_gidx_value) {
      gidx_buffer_writer.AtomicWriteSymbol(gidx_buffer.data(), com_ctx.bin_,
        com_ctx.row_offset_in_batch_ - com_ctx.base_item_offset_ +
        com_ctx.total_items_processed_ + com_ctx.ifeature_);

      // TODO(sriramch): There may be multiple writes to the row_buffer at irow + base_row
      // It should be harmless, as the writes are atomic. Explore if there is a way to avoid it,
      // as the atomic ops are needless after the first write
      gidx_row_writer.AtomicWriteSymbol(gidx_row_buffer.data(),
        com_ctx.row_offset_in_batch_ - com_ctx.base_item_offset_ + com_ctx.total_items_processed_,
        (com_ctx.irow_ + com_ctx.base_row_));

      // Write to the last element of the row index containing total number of items
      if (com_ctx.irow_ + com_ctx.base_row_ + 1 == n_rows) {
        gidx_row_writer.AtomicWriteSymbol(gidx_row_buffer.data(), n_items, n_rows);
      }
    }
  }

  __device__  explicit CSRMatrix(
    common::Span<uint32_t> fsegs, common::Span<bst_float> min_fvals,
    common::Span<bst_float> fval_map, common::CompressedBufferWriter buf_wr,
    common::CompressedIterator<uint32_t> buf_itr, common::Span<common::CompressedByteT> buf,
    int ngidx, size_t rstride, common::CompressedBufferWriter row_wr,
    common::CompressedIterator<uint32_t> row_itr, common::Span<common::CompressedByteT> row_buf,
    size_t nrows, size_t nitems)
      : MatrixBase(fsegs, min_fvals, fval_map, buf_wr, buf_itr, buf, ngidx, rstride),
        gidx_row_writer(row_wr), gidx_row_iter(row_itr), gidx_row_buffer(row_buf),
        n_rows(nrows), n_items(nitems) {}
};

template <typename BaseType, typename DerivedType, typename... Args>
__global__ void DeviceMatrixTypeCreatorKernel(BaseType **obj, Args... args) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *obj = new DerivedType(args...);
  }
}

template<typename std::enable_if<true,  int>::type = 0>
__global__ void DeviceMatrixTypeDestroyerKernel(MatrixBase **ptr) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    delete *ptr;
  }
}

/** \brief Struct for accessing and manipulating an ellpack matrix on the
 * device. The underlying matrix is created on the device and shared by multiple
 * instances. That has to be deleted before the instance itself is destructed
 * to avoid leaks.
 */
struct ELLPackMatrix {
  __forceinline__ __device__ size_t BinCount() const { return (*matrix)->BinCount(); }
  __forceinline__ __device__ size_t RowStride() const { return (*matrix)->RowStride(); }
  __forceinline__ __device__ uint32_t GetFeatureBin(int fidx) const {
    return (*matrix)->GetFeatureBin(fidx);
  }
  __forceinline__ __device__ bst_float GetMinFeatureValue(int fidx) const {
    return (*matrix)->GetMinFeatureValue(fidx);
  }
  __forceinline__ __device__ const bst_float *GetFeatureValue(int fbin) const {
    return (*matrix)->GetFeatureValue(fbin);
  }
  __forceinline__ __device__ int NullGidxValue() const { return (*matrix)->NullGidxValue(); }

  bool is_dense;  // Is the matrix dense? Kept here for tests
  CompressedDataLayout data_layout;  // Kept here for tests
  MatrixBase **matrix;  // Base matrix reference that can be handled polymorphically

  // Get a matrix element, uses binary search for look up
  // Return NaN if missing
  __forceinline__ __device__ bst_float GetElement(size_t ridx, size_t fidx) const {
    return (*matrix)->GetElement(ridx, fidx);
  }

  // Get the gidx value for row ridx and the feature at the gidx_pos in the gidx_buffer
  __forceinline__ __device__ int GetGidx(size_t ridx, size_t gidx_pos) const {
    return (*matrix)->GetGidx(ridx, gidx_pos);
  }

  __forceinline__ __device__ void Write(const CompressRowContext &com_ctx) {
    return (*matrix)->Write(com_ctx);
  }

  ELLPackMatrix(
    common::Span<uint32_t> feature_segments,
    common::Span<bst_float> min_fvalue,
    common::Span<bst_float> gidx_fvalue_map,
    common::CompressedBufferWriter buf_wr,
    common::CompressedIterator<uint32_t> buf_iter,
    common::Span<common::CompressedByteT> buf,
    common::CompressedBufferWriter row_wr,
    common::CompressedIterator<uint32_t> row_iter,
    common::Span<common::CompressedByteT> row_buf,
    size_t row_stride,
    bool is_dense,
    int null_gidx_value,
    size_t n_rows,
    size_t n_items,
    CompressedDataLayout data_layout) {
      // Allocate memory for the base type pointer on device
      dh::safe_cuda(hipMalloc(&matrix, sizeof(MatrixBase **)));

      if (is_dense) {
        DeviceMatrixTypeCreatorKernel<MatrixBase, DenseMatrix><<<1, 1>>>(
          matrix, feature_segments, min_fvalue, gidx_fvalue_map,
          buf_wr, buf_iter, buf, null_gidx_value, row_stride);
      } else if (data_layout == CompressedDataLayout::kRowStride) {
        DeviceMatrixTypeCreatorKernel<MatrixBase, RowStrideMatrix><<<1, 1>>>(
          matrix, feature_segments, min_fvalue, gidx_fvalue_map,
          buf_wr, buf_iter, buf, null_gidx_value, row_stride);
      } else if (data_layout == CompressedDataLayout::kCSR) {
        DeviceMatrixTypeCreatorKernel<MatrixBase, CSRMatrix><<<1, 1>>>(
          matrix, feature_segments, min_fvalue, gidx_fvalue_map,
          buf_wr, buf_iter, buf, null_gidx_value, row_stride, row_wr, row_iter, row_buf,
          n_rows, n_items);
      }

      this->is_dense = is_dense;
      this->data_layout = data_layout;
  }
};

struct DeviceMatrixTypeDestroyer {
  void operator()(ELLPackMatrix *ellpack) {
    DeviceMatrixTypeDestroyerKernel<<<1, 1>>>(ellpack->matrix);
    dh::safe_cuda(hipFree(ellpack->matrix));
    delete ellpack;
  }
};

// With constraints
template <typename GradientPairT>
XGBOOST_DEVICE float inline LossChangeMissing(
    const GradientPairT& scan, const GradientPairT& missing, const GradientPairT& parent_sum,
    const float& parent_gain, const GPUTrainingParam& param, int constraint,
    const ValueConstraint& value_constraint,
    bool& missing_left_out) {  // NOLINT
  float missing_left_gain = value_constraint.CalcSplitGain(
      param, constraint, GradStats(scan + missing),
      GradStats(parent_sum - (scan + missing)));
  float missing_right_gain = value_constraint.CalcSplitGain(
      param, constraint, GradStats(scan), GradStats(parent_sum - scan));

  if (missing_left_gain >= missing_right_gain) {
    missing_left_out = true;
    return missing_left_gain - parent_gain;
  } else {
    missing_left_out = false;
    return missing_right_gain - parent_gain;
  }
}

/*!
 * \brief
 *
 * \tparam ReduceT     BlockReduce Type.
 * \tparam TempStorage Cub Shared memory
 *
 * \param begin
 * \param end
 * \param temp_storage Shared memory for intermediate result.
 */
template <int BLOCK_THREADS, typename ReduceT, typename TempStorageT, typename GradientSumT>
__device__ GradientSumT ReduceFeature(common::Span<const GradientSumT> feature_histogram,
                                      TempStorageT* temp_storage) {
  __shared__ hipcub::Uninitialized<GradientSumT> uninitialized_sum;
  GradientSumT& shared_sum = uninitialized_sum.Alias();

  GradientSumT local_sum = GradientSumT();
  // For loop sums features into one block size
  auto begin = feature_histogram.data();
  auto end = begin + feature_histogram.size();
  for (auto itr = begin; itr < end; itr += BLOCK_THREADS) {
    bool thread_active = itr + threadIdx.x < end;
    // Scan histogram
    GradientSumT bin = thread_active ? *(itr + threadIdx.x) : GradientSumT();
    local_sum += bin;
  }
  local_sum = ReduceT(temp_storage->sum_reduce).Reduce(local_sum, hipcub::Sum());
  // Reduction result is stored in thread 0.
  if (threadIdx.x == 0) {
    shared_sum = local_sum;
  }
  __syncthreads();
  return shared_sum;
}

/*! \brief Find the thread with best gain. */
template <int BLOCK_THREADS, typename ReduceT, typename ScanT,
          typename MaxReduceT, typename TempStorageT, typename GradientSumT>
__device__ void EvaluateFeature(
    int fidx, common::Span<const GradientSumT> node_histogram,
    const ELLPackMatrix& matrix,
    DeviceSplitCandidate* best_split,  // shared memory storing best split
    const DeviceNodeStats& node, const GPUTrainingParam& param,
    TempStorageT* temp_storage,  // temp memory for cub operations
    int constraint,              // monotonic_constraints
    const ValueConstraint& value_constraint) {
  // Use pointer from cut to indicate begin and end of bins for each feature.
  uint32_t gidx_begin = matrix.GetFeatureBin(fidx);  // begining bin
  uint32_t gidx_end = matrix.GetFeatureBin(fidx + 1);  // end bin for i^th feature

  // Sum histogram bins for current feature
  GradientSumT const feature_sum = ReduceFeature<BLOCK_THREADS, ReduceT>(
      node_histogram.subspan(gidx_begin, gidx_end - gidx_begin), temp_storage);

  GradientSumT const parent_sum = GradientSumT(node.sum_gradients);
  GradientSumT const missing = parent_sum - feature_sum;
  float const null_gain = -std::numeric_limits<bst_float>::infinity();

  SumCallbackOp<GradientSumT> prefix_op =
      SumCallbackOp<GradientSumT>();
  for (int scan_begin = gidx_begin; scan_begin < gidx_end;
       scan_begin += BLOCK_THREADS) {
    bool thread_active = (scan_begin + threadIdx.x) < gidx_end;

    // Gradient value for current bin.
    GradientSumT bin =
        thread_active ? node_histogram[scan_begin + threadIdx.x] : GradientSumT();
    ScanT(temp_storage->scan).ExclusiveScan(bin, bin, hipcub::Sum(), prefix_op);

    // Whether the gradient of missing values is put to the left side.
    bool missing_left = true;
    float gain = null_gain;
    if (thread_active) {
      gain = LossChangeMissing(bin, missing, parent_sum, node.root_gain, param,
                               constraint, value_constraint, missing_left);
    }

    __syncthreads();

    // Find thread with best gain
    hipcub::KeyValuePair<int, float> tuple(threadIdx.x, gain);
    hipcub::KeyValuePair<int, float> best =
        MaxReduceT(temp_storage->max_reduce).Reduce(tuple, hipcub::ArgMax());

    __shared__ hipcub::KeyValuePair<int, float> block_max;
    if (threadIdx.x == 0) {
      block_max = best;
    }

    __syncthreads();

    // Best thread updates split
    if (threadIdx.x == block_max.key) {
      int split_gidx = (scan_begin + threadIdx.x) - 1;
      float fvalue;
      if (split_gidx < static_cast<int>(gidx_begin)) {
        fvalue =  matrix.GetMinFeatureValue(fidx);
      } else {
        fvalue = *(matrix.GetFeatureValue(split_gidx));
      }
      GradientSumT left = missing_left ? bin + missing : bin;
      GradientSumT right = parent_sum - left;
      best_split->Update(gain, missing_left ? kLeftDir : kRightDir, fvalue,
                         fidx, GradientPair(left), GradientPair(right), param);
    }
    __syncthreads();
  }
}

template <int BLOCK_THREADS, typename GradientSumT>
__global__ void EvaluateSplitKernel(
    common::Span<const GradientSumT> node_histogram,  // histogram for gradients
    common::Span<const int> feature_set,              // Selected features
    DeviceNodeStats node,
    ELLPackMatrix matrix,
    GPUTrainingParam gpu_param,
    common::Span<DeviceSplitCandidate> split_candidates,  // resulting split
    ValueConstraint value_constraint,
    common::Span<int> d_monotonic_constraints) {
  // KeyValuePair here used as threadIdx.x -> gain_value
  using ArgMaxT = hipcub::KeyValuePair<int, float>;
  using BlockScanT =
      hipcub::BlockScan<GradientSumT, BLOCK_THREADS, hipcub::BLOCK_SCAN_WARP_SCANS>;
  using MaxReduceT = hipcub::BlockReduce<ArgMaxT, BLOCK_THREADS>;

  using SumReduceT = hipcub::BlockReduce<GradientSumT, BLOCK_THREADS>;

  union TempStorage {
    typename BlockScanT::TempStorage scan;
    typename MaxReduceT::TempStorage max_reduce;
    typename SumReduceT::TempStorage sum_reduce;
  };

  // Aligned && shared storage for best_split
  __shared__ hipcub::Uninitialized<DeviceSplitCandidate> uninitialized_split;
  DeviceSplitCandidate& best_split = uninitialized_split.Alias();
  __shared__ TempStorage temp_storage;

  if (threadIdx.x == 0) {
    best_split = DeviceSplitCandidate();
  }

  __syncthreads();

  // One block for each feature. Features are sampled, so fidx != blockIdx.x
  int fidx = feature_set[blockIdx.x];

  int constraint = d_monotonic_constraints[fidx];
  EvaluateFeature<BLOCK_THREADS, SumReduceT, BlockScanT, MaxReduceT>(
      fidx, node_histogram, matrix, &best_split, node, gpu_param, &temp_storage,
      constraint, value_constraint);

  __syncthreads();

  if (threadIdx.x == 0) {
    // Record best loss for each feature
    split_candidates[blockIdx.x] = best_split;
  }
}

/**
 * \struct  DeviceHistogram
 *
 * \summary Data storage for node histograms on device. Automatically expands.
 *
 * \tparam GradientSumT      histogram entry type.
 * \tparam kStopGrowingSize  Do not grow beyond this size
 *
 * \author  Rory
 * \date    28/07/2018
 */
template <typename GradientSumT, size_t kStopGrowingSize = 1 << 26>
class DeviceHistogram {
 private:
  /*! \brief Map nidx to starting index of its histogram. */
  std::map<int, size_t> nidx_map_;
  dh::device_vector<typename GradientSumT::ValueT> data_;
  int n_bins_;
  int device_id_;
  static constexpr size_t kNumItemsInGradientSum =
      sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT);
  static_assert(kNumItemsInGradientSum == 2,
                "Number of items in gradient type should be 2.");

 public:
  void Init(int device_id, int n_bins) {
    this->n_bins_ = n_bins;
    this->device_id_ = device_id;
  }

  void Reset() {
    dh::safe_cuda(hipMemsetAsync(
        data_.data().get(), 0,
        data_.size() * sizeof(typename decltype(data_)::value_type)));
    nidx_map_.clear();
  }
  bool HistogramExists(int nidx) const {
    return nidx_map_.find(nidx) != nidx_map_.cend();
  }
  size_t HistogramSize() const {
    return n_bins_ * kNumItemsInGradientSum;
  }

  dh::device_vector<typename GradientSumT::ValueT>& Data() {
    return data_;
  }

  void AllocateHistogram(int nidx) {
    if (HistogramExists(nidx)) return;
    // Number of items currently used in data
    const size_t used_size = nidx_map_.size() * HistogramSize();
    const size_t new_used_size = used_size + HistogramSize();
    dh::safe_cuda(hipSetDevice(device_id_));
    if (data_.size() >= kStopGrowingSize) {
      // Recycle histogram memory
      if (new_used_size <= data_.size()) {
        // no need to remove old node, just insert the new one.
        nidx_map_[nidx] = used_size;
        // memset histogram size in bytes
        dh::safe_cuda(hipMemsetAsync(data_.data().get() + used_size, 0,
                                      n_bins_ * sizeof(GradientSumT)));
      } else {
        std::pair<int, size_t> old_entry = *nidx_map_.begin();
        nidx_map_.erase(old_entry.first);
        dh::safe_cuda(hipMemsetAsync(data_.data().get() + old_entry.second, 0,
                                      n_bins_ * sizeof(GradientSumT)));
        nidx_map_[nidx] = old_entry.second;
      }
    } else {
      // Append new node histogram
      nidx_map_[nidx] = used_size;
      size_t new_required_memory = std::max(data_.size() * 2, HistogramSize());
      if (data_.size() < new_required_memory) {
        data_.resize(new_required_memory);
      }
    }
  }

  /**
   * \summary   Return pointer to histogram memory for a given node.
   * \param nidx    Tree node index.
   * \return    hist pointer.
   */
  common::Span<GradientSumT> GetNodeHistogram(int nidx) {
    CHECK(this->HistogramExists(nidx));
    auto ptr = data_.data().get() + nidx_map_[nidx];
    return common::Span<GradientSumT>(
        reinterpret_cast<GradientSumT*>(ptr), n_bins_);
  }
};

struct CalcWeightTrainParam {
  float min_child_weight;
  float reg_alpha;
  float reg_lambda;
  float max_delta_step;
  float learning_rate;
  XGBOOST_DEVICE explicit CalcWeightTrainParam(const TrainParam& p)
      : min_child_weight(p.min_child_weight),
        reg_alpha(p.reg_alpha),
        reg_lambda(p.reg_lambda),
        max_delta_step(p.max_delta_step),
        learning_rate(p.learning_rate) {}
};

// Bin each input data entry, store the bin indices in compressed form.
template<typename std::enable_if<true,  int>::type = 0>
__global__ void CompressBinEllpackKernel(
    ELLPackMatrix matrix,
    const size_t* __restrict__ row_ptrs,    // row offset of input data
    const Entry* __restrict__ entries,      // One batch of input data
    size_t base_row,                        // batch_row_begin
    size_t batch_nrows,                     // number of rows in the batch
    size_t base_item_offset,                // item offset from the beginning of the batch
    size_t total_items_processed            // Number of row items processed in the previous batch
    ) {
  size_t irow = threadIdx.x + blockIdx.x * blockDim.x;
  int ifeature = threadIdx.y + blockIdx.y * blockDim.y;
  if (irow >= batch_nrows || ifeature >= matrix.RowStride()) {
    return;
  }
  int row_length = static_cast<int>(row_ptrs[irow + 1] - row_ptrs[irow]);
  unsigned int bin = matrix.NullGidxValue();
  if (ifeature < row_length) {
    Entry entry = entries[row_ptrs[irow] - row_ptrs[0] + ifeature];
    int feature = entry.index;
    float fvalue = entry.fvalue;
    // {feature_cuts, ncuts} forms the array of cuts of `feature'.
    const float *feature_cuts = matrix.GetFeatureValue(matrix.GetFeatureBin(feature));
    int ncuts = matrix.GetFeatureBin(feature + 1) - matrix.GetFeatureBin(feature);
    // Assigning the bin in current entry.
    // S.t.: fvalue < feature_cuts[bin]
    bin = dh::UpperBound(feature_cuts, ncuts, fvalue);
    if (bin >= ncuts) {
      bin = ncuts - 1;
    }
    // Add the number of bins in previous features.
    bin += matrix.GetFeatureBin(feature);
  }

  // Write to gidx buffer.
  CompressRowContext comp_row_ctx(
    bin, irow, base_row, row_ptrs[irow], ifeature, base_item_offset, total_items_processed);
  matrix.Write(comp_row_ctx);
}

template <typename GradientSumT>
__global__ void SharedMemHistKernel(ELLPackMatrix matrix,
                                    common::Span<const RowPartitioner::RowIndexT> d_ridx,
                                    GradientSumT* d_node_hist,
                                    const GradientPair* d_gpair, size_t n_elements,
                                    bool use_shared_memory_histograms) {
  extern __shared__ char smem[];
  GradientSumT* smem_arr = reinterpret_cast<GradientSumT*>(smem);  // NOLINT
  if (use_shared_memory_histograms) {
    dh::BlockFill(smem_arr, matrix.BinCount(), GradientSumT());
    __syncthreads();
  }
  for (auto idx : dh::GridStrideRange(static_cast<size_t>(0), n_elements)) {
    int ridx = d_ridx[idx / matrix.RowStride()];
    int gidx = matrix.GetGidx(ridx, idx);
    if (gidx != matrix.NullGidxValue()) {
      // If we are not using shared memory, accumulate the values directly into
      // global memory
      GradientSumT* atomic_add_ptr =
          use_shared_memory_histograms ? smem_arr : d_node_hist;
      AtomicAddGpair(atomic_add_ptr + gidx, d_gpair[ridx]);
    }
  }

  if (use_shared_memory_histograms) {
    // Write shared memory back to global memory
    __syncthreads();
    for (auto i :
         dh::BlockStrideRange(static_cast<size_t>(0), matrix.BinCount())) {
      AtomicAddGpair(d_node_hist + i, smem_arr[i]);
    }
  }
}

// Instances of this type are created while creating the histogram bins for the
// entire dataset across multiple sparse page batches. This keeps track of the number
// of rows to process from a batch and the position from which to process on each device.
struct RowStateOnDevice {
  // Number of rows assigned to this device
  const size_t total_rows_assigned_to_device;
  // Number of rows processed thus far
  size_t total_rows_processed;
  // Number of rows to process from the current sparse page batch
  size_t rows_to_process_from_batch;
  // Offset from the current sparse page batch to begin processing
  size_t row_offset_in_current_batch;
  // Total number of items processed thus far
  size_t total_items_processed;

  explicit RowStateOnDevice(size_t total_rows)
    : total_rows_assigned_to_device(total_rows), total_rows_processed(0),
      rows_to_process_from_batch(0), row_offset_in_current_batch(0),
      total_items_processed(0) {
  }

  explicit RowStateOnDevice(size_t total_rows, size_t batch_rows)
    : total_rows_assigned_to_device(total_rows), total_rows_processed(0),
      rows_to_process_from_batch(batch_rows), row_offset_in_current_batch(0),
      total_items_processed(0) {
  }

  // Advance the row state by the number of rows processed
  void Advance(const SparsePage &batch) {
    if (rows_to_process_from_batch) {
      const auto &offset_vec = batch.offset.ConstHostVector();
      total_items_processed += offset_vec[row_offset_in_current_batch + rows_to_process_from_batch]
                               - offset_vec[row_offset_in_current_batch];
    }
    total_rows_processed += rows_to_process_from_batch;
    CHECK_LE(total_rows_processed, total_rows_assigned_to_device);
    rows_to_process_from_batch = row_offset_in_current_batch = 0;
  }
};

// Manage memory for a single GPU
template <typename GradientSumT>
struct DeviceShard {
  int device_id;
  int shard_idx;  // Position in the local array of shards

  dh::BulkAllocator ba;

  std::unique_ptr<ELLPackMatrix, DeviceMatrixTypeDestroyer> ellpack_matrix;

  std::unique_ptr<RowPartitioner> row_partitioner;
  DeviceHistogram<GradientSumT> hist;

  /*! \brief row_ptr form HistogramCuts. */
  common::Span<uint32_t> feature_segments;
  /*! \brief minimum value for each feature. */
  common::Span<bst_float> min_fvalue;
  /*! \brief Cut. */
  common::Span<bst_float> gidx_fvalue_map;
  /*! \brief global index of histogram, which is stored in ELLPack format. */
  common::Span<common::CompressedByteT> gidx_buffer;
  /*! \brief for sparse matrices, where an alternate representation can save memory, this
      contains the row indices for the different entries present in gidx_buffer */
  common::Span<common::CompressedByteT> gidx_row_buffer;

  /*! \brief Gradient pair for each row. */
  common::Span<GradientPair> gpair;

  common::Span<int> monotone_constraints;
  common::Span<bst_float> prediction_cache;

  /*! \brief Sum gradient for each node. */
  std::vector<GradientPair> node_sum_gradients;
  common::Span<GradientPair> node_sum_gradients_d;
  /*! The row offset for this shard. */
  bst_uint row_begin_idx;
  bst_uint row_end_idx;
  bst_uint n_rows;
  bst_uint n_items;  // Number of items assigned to this shard
  size_t row_stride;
  int n_bins;

  TrainParam param;
  bool prediction_cache_initialised;
  bool use_shared_memory_histograms {false};

  dh::CubMemory temp_memory;
  dh::PinnedMemory pinned_memory;

  std::vector<hipStream_t> streams;

  common::Monitor monitor;
  std::vector<ValueConstraint> node_value_constraints;
  common::ColumnSampler column_sampler;
  FeatureInteractionConstraint interaction_constraints;

  using ExpandQueue =
      std::priority_queue<ExpandEntry, std::vector<ExpandEntry>,
                          std::function<bool(ExpandEntry, ExpandEntry)>>;
  std::unique_ptr<ExpandQueue> qexpand;

  DeviceShard(int _device_id, int shard_idx, bst_uint row_begin,
              bst_uint row_end, TrainParam _param,
              uint32_t column_sampler_seed,
              uint32_t n_features)
      : device_id(_device_id),
        shard_idx(shard_idx),
        row_begin_idx(row_begin),
        row_end_idx(row_end),
        n_rows(row_end - row_begin),
        n_items(0),
        row_stride(0),
        n_bins(0),
        param(std::move(_param)),
        prediction_cache_initialised(false),
        column_sampler(column_sampler_seed),
        interaction_constraints(param, n_features) {
    monitor.Init(std::string("DeviceShard") + std::to_string(device_id));
  }

  void ComputeItemsInShard(const SparsePage &row_batch, const RowStateOnDevice &device_row_state);

  void InitCompressedData(
      const common::HistogramCuts& hmat, size_t row_stride, bool is_dense);

  void CreateHistIndices(
      const SparsePage &row_batch, const common::HistogramCuts &hmat,
      const RowStateOnDevice &device_row_state, int rows_per_batch);

  ~DeviceShard() {
    dh::safe_cuda(hipSetDevice(device_id));
    for (auto& stream : streams) {
      dh::safe_cuda(hipStreamDestroy(stream));
    }
  }

  // Get vector of at least n initialised streams
  std::vector<hipStream_t>& GetStreams(int n) {
    if (n > streams.size()) {
      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamDestroy(stream));
      }

      streams.clear();
      streams.resize(n);

      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamCreate(&stream));
      }
    }

    return streams;
  }

  // Reset values for each update iteration
  // Note that the column sampler must be passed by value because it is not
  // thread safe
  void Reset(HostDeviceVector<GradientPair>* dh_gpair, int64_t num_columns) {
    if (param.grow_policy == TrainParam::kLossGuide) {
      qexpand.reset(new ExpandQueue(LossGuide));
    } else {
      qexpand.reset(new ExpandQueue(DepthWise));
    }
    this->column_sampler.Init(num_columns, param.colsample_bynode,
      param.colsample_bylevel, param.colsample_bytree);
    dh::safe_cuda(hipSetDevice(device_id));
    this->interaction_constraints.Reset();
    std::fill(node_sum_gradients.begin(), node_sum_gradients.end(),
              GradientPair());
    row_partitioner.reset();  // Release the device memory first before reallocating
    row_partitioner.reset(new RowPartitioner(device_id, n_rows));

    gpair = dh_gpair->DeviceSpan(device_id);

    SubsampleGradientPair(device_id, gpair, param.subsample, row_begin_idx);
    hist.Reset();
  }

  std::vector<DeviceSplitCandidate> EvaluateSplits(
      std::vector<int> nidxs, const RegTree& tree,
      size_t num_columns) {
    dh::safe_cuda(hipSetDevice(device_id));
    auto result_all = pinned_memory.GetSpan<DeviceSplitCandidate>(nidxs.size());

    // Work out cub temporary memory requirement
    GPUTrainingParam gpu_param(param);
    DeviceSplitCandidateReduceOp op(gpu_param);
    size_t temp_storage_bytes = 0;
    DeviceSplitCandidate*dummy = nullptr;
    hipcub::DeviceReduce::Reduce(
        nullptr, temp_storage_bytes, dummy,
        dummy, num_columns, op,
        DeviceSplitCandidate());
    // size in terms of DeviceSplitCandidate
    size_t cub_memory_size =
      std::ceil(static_cast<double>(temp_storage_bytes) /
        sizeof(DeviceSplitCandidate));

    // Allocate enough temporary memory
    // Result for each nidx
    // + intermediate result for each column
    // + cub reduce memory
    auto temp_span = temp_memory.GetSpan<DeviceSplitCandidate>(
        nidxs.size() + nidxs.size() * num_columns +cub_memory_size*nidxs.size());
    auto d_result_all = temp_span.subspan(0, nidxs.size());
    auto d_split_candidates_all =
        temp_span.subspan(d_result_all.size(), nidxs.size() * num_columns);
    auto d_cub_memory_all =
        temp_span.subspan(d_result_all.size() + d_split_candidates_all.size(),
                          cub_memory_size * nidxs.size());

    auto& streams = this->GetStreams(nidxs.size());
    for (auto i = 0ull; i < nidxs.size(); i++) {
      auto nidx = nidxs[i];
      auto p_feature_set = column_sampler.GetFeatureSet(tree.GetDepth(nidx));
      p_feature_set->Shard(GPUSet(device_id, 1));
      auto d_sampled_features = p_feature_set->DeviceSpan(device_id);
      common::Span<int32_t> d_feature_set =
          interaction_constraints.Query(d_sampled_features, nidx);
      auto d_split_candidates =
          d_split_candidates_all.subspan(i * num_columns, d_feature_set.size());

      DeviceNodeStats node(node_sum_gradients[nidx], nidx, param);

      auto d_result = d_result_all.subspan(i, 1);
      if (d_feature_set.size() == 0) {
        // Acting as a device side constructor for DeviceSplitCandidate.
        // DeviceSplitCandidate::IsValid is false so that ApplySplit can reject this
        // candidate.
        auto worst_candidate = DeviceSplitCandidate();
        dh::safe_cuda(hipMemcpyAsync(d_result.data(), &worst_candidate,
                                      sizeof(DeviceSplitCandidate),
                                      hipMemcpyHostToDevice));
        continue;
      }

      // One block for each feature
      int constexpr kBlockThreads = 256;
      EvaluateSplitKernel<kBlockThreads, GradientSumT>
          <<<uint32_t(d_feature_set.size()), kBlockThreads, 0, streams[i]>>>(
              hist.GetNodeHistogram(nidx), d_feature_set, node, *ellpack_matrix,
              gpu_param, d_split_candidates, node_value_constraints[nidx],
              monotone_constraints);

      // Reduce over features to find best feature
      auto d_cub_memory =
          d_cub_memory_all.subspan(i * cub_memory_size, cub_memory_size);
      size_t cub_bytes = d_cub_memory.size() * sizeof(DeviceSplitCandidate);
      hipcub::DeviceReduce::Reduce(reinterpret_cast<void*>(d_cub_memory.data()),
                                cub_bytes, d_split_candidates.data(),
                                d_result.data(), d_split_candidates.size(), op,
                                DeviceSplitCandidate(), streams[i]);
    }

    dh::safe_cuda(hipMemcpy(result_all.data(), d_result_all.data(),
                             sizeof(DeviceSplitCandidate) * d_result_all.size(),
                             hipMemcpyDeviceToHost));
    return std::vector<DeviceSplitCandidate>(result_all.begin(), result_all.end());
  }

  void BuildHist(int nidx) {
    hist.AllocateHistogram(nidx);
    auto d_node_hist = hist.GetNodeHistogram(nidx);

    auto d_ridx = row_partitioner->GetRows(nidx);
    if (!d_ridx.size()) return;

    auto d_gpair = gpair.data();

    auto n_elements = d_ridx.size() * row_stride;

    const size_t smem_size =
        use_shared_memory_histograms
            ? sizeof(GradientSumT) * gidx_fvalue_map.size()
            : 0;
    const int items_per_thread = 8;
    const int block_threads = 256;
    const int grid_size = static_cast<int>(
        common::DivRoundUp(n_elements, items_per_thread * block_threads));
    SharedMemHistKernel<<<grid_size, block_threads, smem_size>>>(
        *ellpack_matrix, d_ridx, d_node_hist.data(), d_gpair, n_elements,
        use_shared_memory_histograms);
  }

  void SubtractionTrick(int nidx_parent, int nidx_histogram,
                        int nidx_subtraction) {
    auto d_node_hist_parent = hist.GetNodeHistogram(nidx_parent);
    auto d_node_hist_histogram = hist.GetNodeHistogram(nidx_histogram);
    auto d_node_hist_subtraction = hist.GetNodeHistogram(nidx_subtraction);

    dh::LaunchN(device_id, n_bins, [=] __device__(size_t idx) {
      d_node_hist_subtraction[idx] =
          d_node_hist_parent[idx] - d_node_hist_histogram[idx];
    });
  }

  bool CanDoSubtractionTrick(int nidx_parent, int nidx_histogram,
                             int nidx_subtraction) {
    // Make sure histograms are already allocated
    hist.AllocateHistogram(nidx_subtraction);
    return hist.HistogramExists(nidx_histogram) &&
           hist.HistogramExists(nidx_parent);
  }

  void UpdatePosition(int nidx, RegTree::Node split_node) {
    auto d_matrix = *ellpack_matrix;

    row_partitioner->UpdatePosition(
        nidx, split_node.LeftChild(), split_node.RightChild(),
        [=] __device__(bst_uint ridx) {
          bst_float element =
              d_matrix.GetElement(ridx, split_node.SplitIndex());
          // Missing value
          int new_position = 0;
          if (isnan(element)) {
            new_position = split_node.DefaultChild();
          } else {
            if (element <= split_node.SplitCond()) {
              new_position = split_node.LeftChild();
            } else {
              new_position = split_node.RightChild();
            }
          }
          return new_position;
        });
  }

  // After tree update is finished, update the position of all training
  // instances to their final leaf This information is used later to update the
  // prediction cache
  void FinalisePosition(RegTree* p_tree) {
    const auto d_nodes =
        temp_memory.GetSpan<RegTree::Node>(p_tree->GetNodes().size());
    dh::safe_cuda(hipMemcpy(d_nodes.data(), p_tree->GetNodes().data(),
                             d_nodes.size() * sizeof(RegTree::Node),
                             hipMemcpyHostToDevice));
    auto d_matrix = *ellpack_matrix;
    row_partitioner->FinalisePosition(
        [=] __device__(bst_uint ridx, int position) {
          auto node = d_nodes[position];

          while (!node.IsLeaf()) {
            bst_float element = d_matrix.GetElement(ridx, node.SplitIndex());
            // Missing value
            if (isnan(element)) {
              position = node.DefaultChild();
            } else {
              if (element <= node.SplitCond()) {
                position = node.LeftChild();
              } else {
                position = node.RightChild();
              }
            }
            node = d_nodes[position];
          }
          return position;
        });
  }

  void UpdatePredictionCache(bst_float* out_preds_d) {
    dh::safe_cuda(hipSetDevice(device_id));
    if (!prediction_cache_initialised) {
      dh::safe_cuda(hipMemcpyAsync(prediction_cache.data(), out_preds_d,
                                    prediction_cache.size() * sizeof(bst_float),
                                    hipMemcpyDefault));
    }
    prediction_cache_initialised = true;

    CalcWeightTrainParam param_d(param);

    dh::safe_cuda(
        hipMemcpyAsync(node_sum_gradients_d.data(), node_sum_gradients.data(),
                        sizeof(GradientPair) * node_sum_gradients.size(),
                        hipMemcpyHostToDevice));
    auto d_position = row_partitioner->GetPosition();
    auto d_ridx = row_partitioner->GetRows();
    auto d_node_sum_gradients = node_sum_gradients_d.data();
    auto d_prediction_cache = prediction_cache.data();

    dh::LaunchN(
        device_id, prediction_cache.size(), [=] __device__(int local_idx) {
          int pos = d_position[local_idx];
          bst_float weight = CalcWeight(param_d, d_node_sum_gradients[pos]);
          d_prediction_cache[d_ridx[local_idx]] +=
              weight * param_d.learning_rate;
        });

    dh::safe_cuda(hipMemcpy(
        out_preds_d, prediction_cache.data(),
        prediction_cache.size() * sizeof(bst_float), hipMemcpyDefault));
    row_partitioner.reset();
  }

  void AllReduceHist(int nidx, dh::AllReducer* reducer) {
    monitor.StartCuda("AllReduce");
    auto d_node_hist = hist.GetNodeHistogram(nidx).data();
    reducer->AllReduceSum(
        shard_idx,
        reinterpret_cast<typename GradientSumT::ValueT*>(d_node_hist),
        reinterpret_cast<typename GradientSumT::ValueT*>(d_node_hist),
        gidx_fvalue_map.size() *
            (sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT)));
    reducer->Synchronize(device_id);

    monitor.StopCuda("AllReduce");
  }

  /**
   * \brief Build GPU local histograms for the left and right child of some parent node
   */
  void BuildHistLeftRight(int nidx_parent, int nidx_left, int nidx_right, dh::AllReducer* reducer) {
    auto build_hist_nidx = nidx_left;
    auto subtraction_trick_nidx = nidx_right;

    auto left_node_rows = row_partitioner->GetRows(nidx_left).size();
    auto right_node_rows = row_partitioner->GetRows(nidx_right).size();
    // Decide whether to build the left histogram or right histogram
    // Find the largest number of training instances on any given Shard
    // Assume this will be the bottleneck and avoid building this node if
    // possible
    std::vector<size_t> max_reduce;
    max_reduce.push_back(left_node_rows);
    max_reduce.push_back(right_node_rows);
    reducer->HostMaxAllReduce(&max_reduce);
    bool fewer_right = max_reduce[1] < max_reduce[0];
    if (fewer_right) {
      std::swap(build_hist_nidx, subtraction_trick_nidx);
    }

    this->BuildHist(build_hist_nidx);
    this->AllReduceHist(build_hist_nidx, reducer);

    // Check whether we can use the subtraction trick to calculate the other
    bool do_subtraction_trick = this->CanDoSubtractionTrick(
        nidx_parent, build_hist_nidx, subtraction_trick_nidx);

    if (do_subtraction_trick) {
      // Calculate other histogram using subtraction trick
      this->SubtractionTrick(nidx_parent, build_hist_nidx,
                             subtraction_trick_nidx);
    } else {
      // Calculate other histogram manually
      this->BuildHist(subtraction_trick_nidx);
      this->AllReduceHist(subtraction_trick_nidx, reducer);
    }
  }
  void ApplySplit(const ExpandEntry& candidate, RegTree* p_tree) {
    RegTree& tree = *p_tree;

    GradStats left_stats;
    left_stats.Add(candidate.split.left_sum);
    GradStats right_stats;
    right_stats.Add(candidate.split.right_sum);
    GradStats parent_sum;
    parent_sum.Add(left_stats);
    parent_sum.Add(right_stats);
    node_value_constraints.resize(tree.GetNodes().size());
    auto base_weight = node_value_constraints[candidate.nid].CalcWeight(param, parent_sum);
    auto left_weight =
        node_value_constraints[candidate.nid].CalcWeight(param, left_stats)*param.learning_rate;
    auto right_weight =
        node_value_constraints[candidate.nid].CalcWeight(param, right_stats)*param.learning_rate;
    tree.ExpandNode(candidate.nid, candidate.split.findex,
                    candidate.split.fvalue, candidate.split.dir == kLeftDir,
                    base_weight, left_weight, right_weight,
                    candidate.split.loss_chg, parent_sum.sum_hess);
    // Set up child constraints
    node_value_constraints.resize(tree.GetNodes().size());
    node_value_constraints[candidate.nid].SetChild(
        param, tree[candidate.nid].SplitIndex(), left_stats, right_stats,
        &node_value_constraints[tree[candidate.nid].LeftChild()],
        &node_value_constraints[tree[candidate.nid].RightChild()]);
    node_sum_gradients[tree[candidate.nid].LeftChild()] =
        candidate.split.left_sum;
    node_sum_gradients[tree[candidate.nid].RightChild()] =
        candidate.split.right_sum;

    interaction_constraints.Split(candidate.nid, tree[candidate.nid].SplitIndex(),
                                  tree[candidate.nid].LeftChild(),
                                  tree[candidate.nid].RightChild());
  }

  void InitRoot(RegTree* p_tree, dh::AllReducer* reducer, int64_t num_columns) {
    constexpr int kRootNIdx = 0;

    reducer->AllReduceSum(
        shard_idx, reinterpret_cast<float*>(node_sum_gradients_d.data()),
        reinterpret_cast<float*>(node_sum_gradients_d.data()), 2);
    reducer->Synchronize(device_id);
    dh::safe_cuda(hipMemcpy(node_sum_gradients.data(),
                             node_sum_gradients_d.data(),
                             sizeof(GradientPair) * node_sum_gradients_d.size(),
                             hipMemcpyDeviceToHost));

    this->BuildHist(kRootNIdx);
    this->AllReduceHist(kRootNIdx, reducer);

    // Remember root stats
    p_tree->Stat(kRootNIdx).sum_hess = node_sum_gradients[kRootNIdx].GetHess();
    auto weight = CalcWeight(param, node_sum_gradients[kRootNIdx]);
    p_tree->Stat(kRootNIdx).base_weight = weight;
    (*p_tree)[kRootNIdx].SetLeaf(param.learning_rate * weight);

    // Initialise root constraint
    node_value_constraints.resize(p_tree->GetNodes().size());

    // Generate first split
    auto split = this->EvaluateSplits({kRootNIdx}, *p_tree, num_columns);
    qexpand->push(
        ExpandEntry(kRootNIdx, p_tree->GetDepth(kRootNIdx), split.at(0), 0));
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair_all, DMatrix* p_fmat,
                  RegTree* p_tree, dh::AllReducer* reducer) {
    auto& tree = *p_tree;

    const auto &gpair_input = gpair_all->DeviceSpan(device_id);
    dh::SumReduction(temp_memory, gpair_input, node_sum_gradients_d, gpair_input.size());

    monitor.StartCuda("Reset");
    this->Reset(gpair_all, p_fmat->Info().num_col_);
    monitor.StopCuda("Reset");

    monitor.StartCuda("InitRoot");
    this->InitRoot(p_tree, reducer, p_fmat->Info().num_col_);
    monitor.StopCuda("InitRoot");
    auto timestamp = qexpand->size();
    auto num_leaves = 1;

    while (!qexpand->empty()) {
      ExpandEntry candidate = qexpand->top();
      qexpand->pop();
      if (!candidate.IsValid(param, num_leaves)) {
        continue;
      }

      this->ApplySplit(candidate, p_tree);

      num_leaves++;

      int left_child_nidx = tree[candidate.nid].LeftChild();
      int right_child_nidx = tree[candidate.nid].RightChild();
      // Only create child entries if needed
      if (ExpandEntry::ChildIsValid(param, tree.GetDepth(left_child_nidx),
                                    num_leaves)) {
        monitor.StartCuda("UpdatePosition");
        this->UpdatePosition(candidate.nid, (*p_tree)[candidate.nid]);
        monitor.StopCuda("UpdatePosition");

        monitor.StartCuda("BuildHist");
        this->BuildHistLeftRight(candidate.nid, left_child_nidx, right_child_nidx, reducer);
        monitor.StopCuda("BuildHist");

        monitor.StartCuda("EvaluateSplits");
        auto splits = this->EvaluateSplits({left_child_nidx, right_child_nidx},
                                           *p_tree, p_fmat->Info().num_col_);
        monitor.StopCuda("EvaluateSplits");

        qexpand->push(ExpandEntry(left_child_nidx,
                                   tree.GetDepth(left_child_nidx), splits.at(0),
                                   timestamp++));
        qexpand->push(ExpandEntry(right_child_nidx,
                                   tree.GetDepth(right_child_nidx),
                                   splits.at(1), timestamp++));
      }
    }

    monitor.StartCuda("FinalisePosition");
    this->FinalisePosition(p_tree);
    monitor.StopCuda("FinalisePosition");
  }
};

template <typename GradientSumT>
inline void DeviceShard<GradientSumT>::ComputeItemsInShard(
  const SparsePage &row_batch,
  const RowStateOnDevice &device_row_state) {
  // Has any been allocated for me in this batch?
  size_t rows_to_process = device_row_state.rows_to_process_from_batch;
  if (!rows_to_process) return;
  const auto &offset_vec = row_batch.offset.ConstHostVector();
  n_items += offset_vec[device_row_state.row_offset_in_current_batch + rows_to_process] -
             + offset_vec[device_row_state.row_offset_in_current_batch];
}

template <typename GradientSumT>
inline void DeviceShard<GradientSumT>::InitCompressedData(
    const common::HistogramCuts &hmat, size_t row_stride, bool is_dense) {
  this->row_stride = row_stride;
  n_bins = hmat.Ptrs().back();
  int null_gidx_value = hmat.Ptrs().back();

  CHECK(!(param.max_leaves == 0 && param.max_depth == 0))
      << "Max leaves and max depth cannot both be unconstrained for "
      "gpu_hist.";

  int max_nodes =
      param.max_leaves > 0 ? param.max_leaves * 2 : MaxNodesDepth(param.max_depth);

  ba.Allocate(device_id,
              &prediction_cache, n_rows,
              &node_sum_gradients_d, max_nodes,
              &feature_segments, hmat.Ptrs().size(),
              &gidx_fvalue_map, hmat.Values().size(),
              &min_fvalue, hmat.MinValues().size(),
              &monotone_constraints, param.monotone_constraints.size());

  dh::CopyVectorToDeviceSpan(gidx_fvalue_map, hmat.Values());
  dh::CopyVectorToDeviceSpan(min_fvalue, hmat.MinValues());
  dh::CopyVectorToDeviceSpan(feature_segments, hmat.Ptrs());
  dh::CopyVectorToDeviceSpan(monotone_constraints, param.monotone_constraints);

  node_sum_gradients.resize(max_nodes);

  // allocate compressed bin data
  int num_symbols = n_bins + 1;
  int num_row_symbols = n_items + 1;

  CompressedDataLayout data_layout = CompressedDataLayout::kRowStride;
  // Required buffer size for storing data matrix in ELLPack format.
  size_t compressed_size_bytes =
    common::CompressedBufferWriter::CalculateBufferSize(row_stride * n_rows,
                                                        num_symbols);
  if (!is_dense) {
    size_t item_compressed_size_bytes =
      common::CompressedBufferWriter::CalculateBufferSize(n_items, num_symbols);

    // +1 for the first element in the row index that contains a value of 0
    size_t row_compressed_size_bytes =
      common::CompressedBufferWriter::CalculateBufferSize(n_rows + 1, num_row_symbols);

    if (item_compressed_size_bytes + row_compressed_size_bytes < compressed_size_bytes) {
      compressed_size_bytes = item_compressed_size_bytes;

      ba.Allocate(device_id, &gidx_row_buffer, row_compressed_size_bytes);
        thrust::fill(
          thrust::device_pointer_cast(gidx_row_buffer.data()),
          thrust::device_pointer_cast(gidx_row_buffer.data() + gidx_row_buffer.size()), 0);

      data_layout = CompressedDataLayout::kCSR;
    }
  }

  ba.Allocate(device_id, &gidx_buffer, compressed_size_bytes);
  thrust::fill(
      thrust::device_pointer_cast(gidx_buffer.data()),
      thrust::device_pointer_cast(gidx_buffer.data() + gidx_buffer.size()), 0);

  ellpack_matrix.reset(
    new ELLPackMatrix(
      feature_segments, min_fvalue, gidx_fvalue_map,
      common::CompressedBufferWriter(num_symbols),
      common::CompressedIterator<uint32_t>(gidx_buffer.data(), num_symbols), gidx_buffer,
      common::CompressedBufferWriter(num_row_symbols),
      common::CompressedIterator<uint32_t>(gidx_row_buffer.data(), num_row_symbols),
      gidx_row_buffer,
      row_stride, is_dense, null_gidx_value, n_rows, n_items, data_layout));

  // check if we can use shared memory for building histograms
  // (assuming atleast we need 2 CTAs per SM to maintain decent latency
  // hiding)
  auto histogram_size = sizeof(GradientSumT) * hmat.Ptrs().back();
  auto max_smem = dh::MaxSharedMemory(device_id);
  if (histogram_size <= max_smem) {
    use_shared_memory_histograms = true;
  }

  // Init histogram
  hist.Init(device_id, hmat.Ptrs().back());
}

template <typename GradientSumT>
inline void DeviceShard<GradientSumT>::CreateHistIndices(
    const SparsePage &row_batch,
    const common::HistogramCuts &hmat,
    const RowStateOnDevice &device_row_state,
    int rows_per_batch) {
  // Has any been allocated for me in this batch?
  if (!device_row_state.rows_to_process_from_batch) return;

  unsigned int null_gidx_value = hmat.Ptrs().back();

  const auto &offset_vec = row_batch.offset.ConstHostVector();
  size_t base_offset = offset_vec[device_row_state.row_offset_in_current_batch];

  // bin and compress entries in batches of rows
  size_t gpu_batch_nrows = std::min(
    dh::TotalMemory(device_id) / (16 * row_stride * sizeof(Entry)),
    static_cast<size_t>(device_row_state.rows_to_process_from_batch));
  const std::vector<Entry>& data_vec = row_batch.data.ConstHostVector();

  size_t gpu_nbatches = common::DivRoundUp(device_row_state.rows_to_process_from_batch,
                                           gpu_batch_nrows);

  for (size_t gpu_batch = 0; gpu_batch < gpu_nbatches; ++gpu_batch) {
    size_t batch_row_begin = gpu_batch * gpu_batch_nrows;
    size_t batch_row_end = (gpu_batch + 1) * gpu_batch_nrows;
    if (batch_row_end > device_row_state.rows_to_process_from_batch) {
      batch_row_end = device_row_state.rows_to_process_from_batch;
    }
    size_t batch_nrows = batch_row_end - batch_row_begin;

    const auto ent_cnt_begin =
      offset_vec[device_row_state.row_offset_in_current_batch + batch_row_begin];
    const auto ent_cnt_end =
      offset_vec[device_row_state.row_offset_in_current_batch + batch_row_end];

    /*! \brief row offset in SparsePage (the input data). */
    dh::device_vector<size_t> row_ptrs(batch_nrows+1);
    thrust::copy(
      offset_vec.data() + device_row_state.row_offset_in_current_batch + batch_row_begin,
      offset_vec.data() + device_row_state.row_offset_in_current_batch + batch_row_end + 1,
      row_ptrs.begin());

    // number of entries in this batch.
    size_t n_entries = ent_cnt_end - ent_cnt_begin;
    dh::device_vector<Entry> entries_d(n_entries);
    // copy data entries to device.
    dh::safe_cuda
        (hipMemcpy
         (entries_d.data().get(), data_vec.data() + ent_cnt_begin,
          n_entries * sizeof(Entry), hipMemcpyDefault));
    const dim3 block3(32, 8, 1);  // 256 threads
    const dim3 grid3(common::DivRoundUp(batch_nrows, block3.x),
                     common::DivRoundUp(row_stride, block3.y), 1);
    CompressBinEllpackKernel<<<grid3, block3>>>
        (*this->ellpack_matrix,
         row_ptrs.data().get(),
         entries_d.data().get(),
         device_row_state.total_rows_processed + batch_row_begin,
         batch_nrows,
         base_offset,
         device_row_state.total_items_processed);
  }
}

// An instance of this type is created which keeps track of total number of rows to process,
// rows processed thus far, rows to process and the offset from the current sparse page batch
// to begin processing on each device
class DeviceHistogramBuilderState {
 public:
  template <typename GradientSumT>
  explicit DeviceHistogramBuilderState(
    const std::vector<std::unique_ptr<DeviceShard<GradientSumT>>> &shards) {
    device_row_states_.reserve(shards.size());
    for (const auto &shard : shards) {
      device_row_states_.push_back(RowStateOnDevice(shard->n_rows));
    }
  }

  const RowStateOnDevice &GetRowStateOnDevice(int idx) const {
    return device_row_states_[idx];
  }

  // This method is invoked at the beginning of each sparse page batch. This distributes
  // the rows in the sparse page to the different devices.
  // TODO(sriramch): Think of a way to utilize *all* the GPUs to build the compressed bins.
  void BeginBatch(const SparsePage &batch) {
    size_t rem_rows = batch.Size();
    size_t row_offset_in_current_batch = 0;
    for (auto &device_row_state : device_row_states_) {
      // Do we have anymore left to process from this batch on this device?
      if (device_row_state.total_rows_assigned_to_device > device_row_state.total_rows_processed) {
        // There are still some rows that needs to be assigned to this device
        device_row_state.rows_to_process_from_batch =
          std::min(
            device_row_state.total_rows_assigned_to_device - device_row_state.total_rows_processed,
            rem_rows);
      } else {
        // All rows have been assigned to this device
        device_row_state.rows_to_process_from_batch = 0;
      }

      device_row_state.row_offset_in_current_batch = row_offset_in_current_batch;
      row_offset_in_current_batch += device_row_state.rows_to_process_from_batch;
      rem_rows -= device_row_state.rows_to_process_from_batch;
    }
  }

  // This method is invoked after completion of each sparse page batch
  void EndBatch(const SparsePage &batch) {
    for (auto &rs : device_row_states_) {
      rs.Advance(batch);
    }
  }

 private:
  std::vector<RowStateOnDevice> device_row_states_;
};

template <typename GradientSumT>
class GPUHistMakerSpecialised {
 public:
  GPUHistMakerSpecialised() : initialised_{false}, p_last_fmat_{nullptr} {}
  void Configure(const Args& args, GenericParameter const* generic_param) {
    param_.InitAllowUnknown(args);
    generic_param_ = generic_param;
    hist_maker_param_.InitAllowUnknown(args);
    auto devices = GPUSet::All(generic_param_->gpu_id,
                               generic_param_->n_gpus);
    n_devices_ = devices.Size();
    CHECK(n_devices_ != 0) << "Must have at least one device";
    dist_ = GPUDistribution::Block(devices);

    dh::CheckComputeCapability();

    monitor_.Init("updater_gpu_hist");
  }

  ~GPUHistMakerSpecialised() { dh::GlobalMemoryLogger().Log(); }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) {
    monitor_.StartCuda("Update");
    // rescale learning rate according to size of trees
    float lr = param_.learning_rate;
    param_.learning_rate = lr / trees.size();
    ValueConstraint::Init(&param_, dmat->Info().num_col_);
    // build tree
    try {
      for (xgboost::RegTree* tree : trees) {
        this->UpdateTree(gpair, dmat, tree);
      }
      dh::safe_cuda(hipGetLastError());
    } catch (const std::exception& e) {
      LOG(FATAL) << "Exception in gpu_hist: " << e.what() << std::endl;
    }
    param_.learning_rate = lr;
    monitor_.StopCuda("Update");
  }

  void InitDataOnce(DMatrix* dmat) {
    info_ = &dmat->Info();

    int n_devices = dist_.Devices().Size();

    device_list_.resize(n_devices);
    for (int index = 0; index < n_devices; ++index) {
      int device_id = dist_.Devices().DeviceId(index);
      device_list_[index] = device_id;
    }

    reducer_.Init(device_list_);

    // Synchronise the column sampling seed
    uint32_t column_sampling_seed = common::GlobalRandom()();
    rabit::Broadcast(&column_sampling_seed, sizeof(column_sampling_seed), 0);

    // Create device shards
    shards_.resize(n_devices);
    dh::ExecuteIndexShards(
        &shards_,
        [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
          dh::safe_cuda(hipSetDevice(dist_.Devices().DeviceId(idx)));
          size_t start = dist_.ShardStart(info_->num_row_, idx);
          size_t size = dist_.ShardSize(info_->num_row_, idx);
          shard = std::unique_ptr<DeviceShard<GradientSumT>>(
            new DeviceShard<GradientSumT>(dist_.Devices().DeviceId(idx), idx,
                                          start, start + size, param_,
                                          column_sampling_seed,
                                          info_->num_col_));
        });

    monitor_.StartCuda("Quantiles");
    // Create the quantile sketches for the dmatrix and initialize HistogramCuts
    size_t row_stride = common::DeviceSketch(param_, *generic_param_,
                                             hist_maker_param_.gpu_batch_nrows,
                                             dmat, &hmat_);
    monitor_.StopCuda("Quantiles");

    n_bins_ = hmat_.Ptrs().back();

    auto is_dense = info_->num_nonzero_ == info_->num_row_ * info_->num_col_;

    // Init global data for each shard
    monitor_.StartCuda("InitCompressedData");
    {
      DeviceHistogramBuilderState hist_builder_row_state(shards_);
      for (const auto &batch : dmat->GetRowBatches()) {
        hist_builder_row_state.BeginBatch(batch);

        dh::ExecuteIndexShards(
          &shards_,
          [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
            dh::safe_cuda(hipSetDevice(shard->device_id));
            shard->ComputeItemsInShard(batch, hist_builder_row_state.GetRowStateOnDevice(idx));
          });

        hist_builder_row_state.EndBatch(batch);
      }
    }

    dh::ExecuteIndexShards(
        &shards_,
        [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
          dh::safe_cuda(hipSetDevice(shard->device_id));
          shard->InitCompressedData(hmat_, row_stride, is_dense);
        });
    monitor_.StopCuda("InitCompressedData");

    monitor_.StartCuda("BinningCompression");
    DeviceHistogramBuilderState hist_builder_row_state(shards_);
    for (const auto &batch : dmat->GetRowBatches()) {
      hist_builder_row_state.BeginBatch(batch);

      dh::ExecuteIndexShards(
        &shards_,
        [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
          dh::safe_cuda(hipSetDevice(shard->device_id));
          shard->CreateHistIndices(batch, hmat_, hist_builder_row_state.GetRowStateOnDevice(idx),
                                   hist_maker_param_.gpu_batch_nrows);
        });

      hist_builder_row_state.EndBatch(batch);
    }
    monitor_.StopCuda("BinningCompression");

    p_last_fmat_ = dmat;
    initialised_ = true;
  }

  void InitData(DMatrix* dmat) {
    if (!initialised_) {
      monitor_.StartCuda("InitDataOnce");
      this->InitDataOnce(dmat);
      monitor_.StopCuda("InitDataOnce");
    }
  }

  // Only call this method for testing
  void CheckTreesSynchronized(const std::vector<RegTree>& local_trees) const {
    std::string s_model;
    common::MemoryBufferStream fs(&s_model);
    int rank = rabit::GetRank();
    if (rank == 0) {
      local_trees.front().Save(&fs);
    }
    fs.Seek(0);
    rabit::Broadcast(&s_model, 0);
    RegTree reference_tree;
    reference_tree.Load(&fs);
    for (const auto& tree : local_trees) {
      CHECK(tree == reference_tree);
    }
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* p_fmat,
                  RegTree* p_tree) {
    monitor_.StartCuda("InitData");
    this->InitData(p_fmat);
    monitor_.StopCuda("InitData");

    std::vector<RegTree> trees(shards_.size());
    for (auto& tree : trees) {
      tree = *p_tree;
    }
    gpair->Reshard(dist_);

    // Launch one thread for each device "shard" containing a subset of rows.
    // Threads will cooperatively build the tree, synchronising over histograms.
    // Each thread will redundantly build its own copy of the tree
    dh::ExecuteIndexShards(
        &shards_,
        [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
          dh::safe_cuda(hipSetDevice(shard->device_id));
          shard->UpdateTree(gpair, p_fmat, &trees.at(idx), &reducer_);
        });

    // All trees are expected to be identical
    if (hist_maker_param_.debug_synchronize) {
      this->CheckTreesSynchronized(trees);
    }

    // Write the output tree
    *p_tree = trees.front();
  }

  bool UpdatePredictionCache(
      const DMatrix* data, HostDeviceVector<bst_float>* p_out_preds) {
    if (shards_.empty() || p_last_fmat_ == nullptr || p_last_fmat_ != data) {
      return false;
    }
    monitor_.StartCuda("UpdatePredictionCache");
    bool cpu_predictor = p_out_preds->Devices().IsEmpty();
    if (!cpu_predictor) p_out_preds->Shard(dist_.Devices());
    dh::ExecuteIndexShards(
        &shards_,
        [&](int idx, std::unique_ptr<DeviceShard<GradientSumT>>& shard) {
          dh::safe_cuda(hipSetDevice(shard->device_id));
          if (cpu_predictor) {
            size_t start = dist_.ShardStart(p_out_preds->Size(), idx);
            shard->UpdatePredictionCache(&(p_out_preds->HostVector())[start]);
          } else {
            shard->UpdatePredictionCache(
              p_out_preds->DevicePointer(shard->device_id));
          }
        });
    monitor_.StopCuda("UpdatePredictionCache");
    return true;
  }

  TrainParam param_;           // NOLINT
  common::HistogramCuts hmat_; // NOLINT
  MetaInfo* info_;             // NOLINT

  std::vector<std::unique_ptr<DeviceShard<GradientSumT>>> shards_;  // NOLINT

 private:
  bool initialised_;

  int n_devices_;
  int n_bins_;

  GPUHistMakerTrainParam hist_maker_param_;
  GenericParameter const* generic_param_;

  dh::AllReducer reducer_;

  DMatrix* p_last_fmat_;
  GPUDistribution dist_;

  common::Monitor monitor_;
  /*! List storing device id. */
  std::vector<int> device_list_;
};

class GPUHistMaker : public TreeUpdater {
 public:
  void Configure(const Args& args) override {
    hist_maker_param_.InitAllowUnknown(args);
    float_maker_.reset();
    double_maker_.reset();
    if (hist_maker_param_.single_precision_histogram) {
      float_maker_.reset(new GPUHistMakerSpecialised<GradientPair>());
      float_maker_->Configure(args, tparam_);
    } else {
      double_maker_.reset(new GPUHistMakerSpecialised<GradientPairPrecise>());
      double_maker_->Configure(args, tparam_);
    }
  }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) override {
    if (hist_maker_param_.single_precision_histogram) {
      float_maker_->Update(gpair, dmat, trees);
    } else {
      double_maker_->Update(gpair, dmat, trees);
    }
  }

  bool UpdatePredictionCache(
      const DMatrix* data, HostDeviceVector<bst_float>* p_out_preds) override {
    if (hist_maker_param_.single_precision_histogram) {
      return float_maker_->UpdatePredictionCache(data, p_out_preds);
    } else {
      return double_maker_->UpdatePredictionCache(data, p_out_preds);
    }
  }

  char const* Name() const override {
    return "gpu_hist";
  }

 private:
  GPUHistMakerTrainParam hist_maker_param_;
  std::unique_ptr<GPUHistMakerSpecialised<GradientPair>> float_maker_;
  std::unique_ptr<GPUHistMakerSpecialised<GradientPairPrecise>> double_maker_;
};

#if !defined(GTEST_TEST)
XGBOOST_REGISTER_TREE_UPDATER(GPUHistMaker, "grow_gpu_hist")
    .describe("Grow tree with GPU.")
    .set_body([]() { return new GPUHistMaker(); });
#endif  // !defined(GTEST_TEST)

}  // namespace tree
}  // namespace xgboost
