#include "hip/hip_runtime.h"
/**
 * Copyright 2017-2024, XGBoost contributors
 */
#include <thrust/functional.h>  // for plus
#include <thrust/transform.h>   // for transform

#include <algorithm>  // for max
#include <cmath>      // for isnan
#include <cstddef>    // for size_t
#include <memory>     // for unique_ptr, make_unique
#include <utility>    // for move
#include <vector>     // for vector

#include "../collective/aggregator.h"
#include "../collective/broadcast.h"   // for Broadcast
#include "../common/categorical.h"     // for KCatBitField
#include "../common/cuda_context.cuh"  // for HIPContext
#include "../common/cuda_rt_utils.h"   // for CheckComputeCapability
#include "../common/device_helpers.cuh"
#include "../common/device_vector.cuh"  // for device_vector
#include "../common/hist_util.h"        // for HistogramCuts
#include "../common/random.h"           // for ColumnSampler, GlobalRandom
#include "../common/timer.h"
#include "../data/ellpack_page.cuh"
#include "../data/ellpack_page.h"
#include "constraints.cuh"
#include "driver.h"
#include "gpu_hist/evaluate_splits.cuh"
#include "gpu_hist/expand_entry.cuh"
#include "gpu_hist/feature_groups.cuh"
#include "gpu_hist/gradient_based_sampler.cuh"
#include "gpu_hist/histogram.cuh"
#include "gpu_hist/row_partitioner.cuh"  // for RowPartitioner
#include "hist/param.h"                  // for HistMakerTrainParam
#include "param.h"                       // for TrainParam
#include "sample_position.h"             // for SamplePosition
#include "updater_gpu_common.cuh"        // for HistBatch
#include "xgboost/base.h"                // for bst_idx_t
#include "xgboost/context.h"             // for Context
#include "xgboost/data.h"                // for DMatrix
#include "xgboost/host_device_vector.h"  // for HostDeviceVector
#include "xgboost/json.h"                // for Json
#include "xgboost/span.h"                // for Span
#include "xgboost/task.h"                // for ObjInfo
#include "xgboost/tree_model.h"          // for RegTree
#include "xgboost/tree_updater.h"        // for TreeUpdater

namespace xgboost::tree {
DMLC_REGISTRY_FILE_TAG(updater_gpu_hist);

using cuda_impl::ApproxBatch;
using cuda_impl::HistBatch;

// Both the approx and hist initializes the DMatrix before creating the actual
// implementation (InitDataOnce). Therefore, the `GPUHistMakerDevice` can use an empty
// parameter to avoid any regen.
using cuda_impl::StaticBatch;

// Extra data for each node that is passed to the update position function
struct NodeSplitData {
  RegTree::Node split_node;
  FeatureType split_type;
  common::KCatBitField node_cats;
};
static_assert(std::is_trivially_copyable_v<NodeSplitData>);

// GPU tree updater implementation.
struct GPUHistMakerDevice {
 private:
  GPUHistEvaluator evaluator_;
  Context const* ctx_;
  std::shared_ptr<common::ColumnSampler> column_sampler_;
  // Set of row partitioners, one for each batch (external memory). When the training is
  // in-core, there's only one partitioner.
  std::vector<std::unique_ptr<RowPartitioner>> partitioners_;

  DeviceHistogramBuilder histogram_;
  std::vector<bst_idx_t> batch_ptr_;
  // node idx for each sample
  dh::device_vector<bst_node_t> positions_;
  std::shared_ptr<common::HistogramCuts const> cuts_{nullptr};

 public:
  DeviceHistogramStorage<> hist{};

  dh::device_vector<GradientPair> d_gpair;  // storage for gpair;
  common::Span<GradientPair const> gpair;

  dh::device_vector<int> monotone_constraints;

  TrainParam param;

  std::unique_ptr<GradientQuantiser> quantiser;

  dh::PinnedMemory pinned;
  dh::PinnedMemory pinned2;

  common::Monitor monitor;
  FeatureInteractionConstraintDevice interaction_constraints;

  std::unique_ptr<GradientBasedSampler> sampler;

  std::unique_ptr<FeatureGroups> feature_groups;

  GPUHistMakerDevice(Context const* ctx, TrainParam _param,
                     std::shared_ptr<common::ColumnSampler> column_sampler, BatchParam batch_param,
                     MetaInfo const& info, std::vector<bst_idx_t> batch_ptr,
                     std::shared_ptr<common::HistogramCuts const> cuts)
      : evaluator_{_param, static_cast<bst_feature_t>(info.num_col_), ctx->Device()},
        ctx_(ctx),
        param(std::move(_param)),
        column_sampler_(std::move(column_sampler)),
        interaction_constraints(param, static_cast<bst_feature_t>(info.num_col_)),
        batch_ptr_{std::move(batch_ptr)},
        cuts_{std::move(cuts)} {
    sampler =
        std::make_unique<GradientBasedSampler>(ctx, info.num_row_, batch_param, param.subsample,
                                               param.sampling_method, batch_ptr_.size() > 2);
    if (!param.monotone_constraints.empty()) {
      // Copy assigning an empty vector causes an exception in MSVC debug builds
      monotone_constraints = param.monotone_constraints;
    }

    CHECK(column_sampler_);
    monitor.Init(std::string("GPUHistMakerDevice") + ctx_->Device().Name());
  }

  ~GPUHistMakerDevice() = default;

  void InitFeatureGroupsOnce(MetaInfo const& info) {
    if (!feature_groups) {
      CHECK(cuts_);
      feature_groups = std::make_unique<FeatureGroups>(*cuts_, info.IsDense(),
                                                       dh::MaxSharedMemoryOptin(ctx_->Ordinal()),
                                                       sizeof(GradientPairPrecise));
    }
  }

  // Reset values for each update iteration
  [[nodiscard]] DMatrix* Reset(HostDeviceVector<GradientPair>* dh_gpair, DMatrix* p_fmat) {
    auto const& info = p_fmat->Info();
    this->column_sampler_->Init(ctx_, p_fmat->Info().num_col_, info.feature_weights.HostVector(),
                                param.colsample_bynode, param.colsample_bylevel,
                                param.colsample_bytree);
    dh::safe_cuda(hipSetDevice(ctx_->Ordinal()));

    this->interaction_constraints.Reset();

    // Sampling
    dh::CopyTo(dh_gpair->ConstDeviceSpan(), &this->d_gpair);  // backup the gradient
    auto sample = this->sampler->Sample(ctx_, dh::ToSpan(d_gpair), p_fmat);
    this->gpair = sample.gpair;
    p_fmat = sample.p_fmat;  // Update p_fmat before allocating partitioners
    p_fmat->Info().feature_types.SetDevice(ctx_->Device());
    std::size_t n_batches = p_fmat->NumBatches();
    bool is_concat = (n_batches + 1) != this->batch_ptr_.size();
    std::vector<bst_idx_t> batch_ptr{batch_ptr_};
    if (is_concat) {
      // Concatenate the batch ptrs as well.
      batch_ptr = {static_cast<bst_idx_t>(0), p_fmat->Info().num_row_};
    }
    // Initialize partitions
    if (!partitioners_.empty()) {
      CHECK_EQ(partitioners_.size(), n_batches);
    }
    for (std::size_t k = 0; k < n_batches; ++k) {
      if (partitioners_.size() != n_batches) {
        // First run.
        partitioners_.emplace_back(std::make_unique<RowPartitioner>());
      }
      auto base_ridx = batch_ptr[k];
      auto n_samples = batch_ptr.at(k + 1) - base_ridx;
      partitioners_[k]->Reset(ctx_, n_samples, base_ridx);
    }
    CHECK_EQ(partitioners_.size(), n_batches);
    if (is_concat) {
      CHECK_EQ(partitioners_.size(), 1);
      CHECK_EQ(partitioners_.front()->Size(), p_fmat->Info().num_row_);
    }

    // Other initializations
    this->evaluator_.Reset(*cuts_, p_fmat->Info().feature_types.ConstDeviceSpan(),
                           p_fmat->Info().num_col_, this->param, p_fmat->Info().IsColumnSplit(),
                           this->ctx_->Device());

    quantiser = std::make_unique<GradientQuantiser>(ctx_, this->gpair, p_fmat->Info());

    // Init histogram
    hist.Init(ctx_->Device(), this->cuts_->TotalBins());
    hist.Reset(ctx_);

    this->InitFeatureGroupsOnce(info);

    this->histogram_.Reset(ctx_, feature_groups->DeviceAccessor(ctx_->Device()), false);
    return p_fmat;
  }

  GPUExpandEntry EvaluateRootSplit(DMatrix const* p_fmat, GradientPairInt64 root_sum) {
    bst_node_t nidx = RegTree::kRoot;
    GPUTrainingParam gpu_param(param);
    auto sampled_features = column_sampler_->GetFeatureSet(0);
    sampled_features->SetDevice(ctx_->Device());
    common::Span<bst_feature_t> feature_set =
        interaction_constraints.Query(sampled_features->DeviceSpan(), nidx);
    EvaluateSplitInputs inputs{nidx, 0, root_sum, feature_set, hist.GetNodeHistogram(nidx)};
    EvaluateSplitSharedInputs shared_inputs{gpu_param,
                                            *quantiser,
                                            p_fmat->Info().feature_types.ConstDeviceSpan(),
                                            cuts_->cut_ptrs_.ConstDeviceSpan(),
                                            cuts_->cut_values_.ConstDeviceSpan(),
                                            cuts_->min_vals_.ConstDeviceSpan(),
                                            p_fmat->IsDense() && !collective::IsDistributed()};
    auto split = this->evaluator_.EvaluateSingleSplit(ctx_, inputs, shared_inputs);
    return split;
  }

  void EvaluateSplits(DMatrix const* p_fmat, const std::vector<GPUExpandEntry>& candidates,
                      const RegTree& tree, common::Span<GPUExpandEntry> pinned_candidates_out) {
    if (candidates.empty()) {
      return;
    }
    this->monitor.Start(__func__);
    dh::TemporaryArray<EvaluateSplitInputs> d_node_inputs(2 * candidates.size());
    dh::TemporaryArray<DeviceSplitCandidate> splits_out(2 * candidates.size());
    std::vector<bst_node_t> nidx(2 * candidates.size());
    auto h_node_inputs = pinned2.GetSpan<EvaluateSplitInputs>(2 * candidates.size());
    EvaluateSplitSharedInputs shared_inputs{
        GPUTrainingParam{param}, *quantiser, p_fmat->Info().feature_types.ConstDeviceSpan(),
        cuts_->cut_ptrs_.ConstDeviceSpan(), cuts_->cut_values_.ConstDeviceSpan(),
        cuts_->min_vals_.ConstDeviceSpan(),
        // is_dense represents the local data
        p_fmat->IsDense() && !collective::IsDistributed()};
    dh::TemporaryArray<GPUExpandEntry> entries(2 * candidates.size());
    // Store the feature set ptrs so they dont go out of scope before the kernel is called
    std::vector<std::shared_ptr<HostDeviceVector<bst_feature_t>>> feature_sets;
    for (std::size_t i = 0; i < candidates.size(); i++) {
      auto candidate = candidates.at(i);
      int left_nidx = tree[candidate.nid].LeftChild();
      int right_nidx = tree[candidate.nid].RightChild();
      nidx[i * 2] = left_nidx;
      nidx[i * 2 + 1] = right_nidx;
      auto left_sampled_features = column_sampler_->GetFeatureSet(tree.GetDepth(left_nidx));
      left_sampled_features->SetDevice(ctx_->Device());
      feature_sets.emplace_back(left_sampled_features);
      common::Span<bst_feature_t> left_feature_set =
          interaction_constraints.Query(left_sampled_features->DeviceSpan(), left_nidx);
      auto right_sampled_features = column_sampler_->GetFeatureSet(tree.GetDepth(right_nidx));
      right_sampled_features->SetDevice(ctx_->Device());
      feature_sets.emplace_back(right_sampled_features);
      common::Span<bst_feature_t> right_feature_set =
          interaction_constraints.Query(right_sampled_features->DeviceSpan(),
                                        right_nidx);
      h_node_inputs[i * 2] = {left_nidx, candidate.depth + 1,
                              candidate.split.left_sum, left_feature_set,
                              hist.GetNodeHistogram(left_nidx)};
      h_node_inputs[i * 2 + 1] = {right_nidx, candidate.depth + 1,
                                  candidate.split.right_sum, right_feature_set,
                                  hist.GetNodeHistogram(right_nidx)};
    }
    bst_feature_t max_active_features = 0;
    for (auto input : h_node_inputs) {
      max_active_features =
          std::max(max_active_features, static_cast<bst_feature_t>(input.feature_set.size()));
    }
    dh::safe_cuda(hipMemcpyAsync(
        d_node_inputs.data().get(), h_node_inputs.data(),
        h_node_inputs.size() * sizeof(EvaluateSplitInputs), hipMemcpyDefault));

    this->evaluator_.EvaluateSplits(ctx_, nidx, max_active_features, dh::ToSpan(d_node_inputs),
                                    shared_inputs, dh::ToSpan(entries));
    dh::safe_cuda(hipMemcpyAsync(pinned_candidates_out.data(),
                                  entries.data().get(), sizeof(GPUExpandEntry) * entries.size(),
                                  hipMemcpyDeviceToHost));
    this->monitor.Stop(__func__);
  }

  void BuildHist(EllpackPageImpl const* page, int nidx) {
    auto d_node_hist = hist.GetNodeHistogram(nidx);
    auto d_ridx = partitioners_.front()->GetRows(nidx);
    this->histogram_.BuildHistogram(ctx_->CUDACtx(), page->GetDeviceAccessor(ctx_->Device()),
                                    feature_groups->DeviceAccessor(ctx_->Device()), gpair, d_ridx,
                                    d_node_hist, *quantiser);
  }

  // Attempt to do subtraction trick
  // return true if succeeded
  bool SubtractionTrick(int nidx_parent, int nidx_histogram, int nidx_subtraction) {
    if (!hist.HistogramExists(nidx_histogram) || !hist.HistogramExists(nidx_parent)) {
      return false;
    }
    auto d_node_hist_parent = hist.GetNodeHistogram(nidx_parent);
    auto d_node_hist_histogram = hist.GetNodeHistogram(nidx_histogram);
    auto d_node_hist_subtraction = hist.GetNodeHistogram(nidx_subtraction);

    dh::LaunchN(cuts_->TotalBins(), [=] __device__(size_t idx) {
      d_node_hist_subtraction[idx] = d_node_hist_parent[idx] - d_node_hist_histogram[idx];
    });
    return true;
  }

  void UpdatePositionColumnSplit(EllpackDeviceAccessor d_matrix,
                                 std::vector<NodeSplitData> const& split_data,
                                 std::vector<bst_node_t> const& nidx,
                                 std::vector<bst_node_t> const& left_nidx,
                                 std::vector<bst_node_t> const& right_nidx) {
    auto const num_candidates = split_data.size();

    using BitVector = LBitField64;
    using BitType = BitVector::value_type;
    auto const size = BitVector::ComputeStorageSize(d_matrix.n_rows * num_candidates);
    dh::TemporaryArray<BitType> decision_storage(size, 0);
    dh::TemporaryArray<BitType> missing_storage(size, 0);
    BitVector decision_bits{dh::ToSpan(decision_storage)};
    BitVector missing_bits{dh::ToSpan(missing_storage)};

    dh::TemporaryArray<NodeSplitData> split_data_storage(num_candidates);
    dh::safe_cuda(hipMemcpyAsync(split_data_storage.data().get(), split_data.data(),
                                  num_candidates * sizeof(NodeSplitData), hipMemcpyDefault));
    auto d_split_data = dh::ToSpan(split_data_storage);

    dh::LaunchN(d_matrix.n_rows, [=] __device__(std::size_t ridx) mutable {
      for (auto i = 0; i < num_candidates; i++) {
        auto const& data = d_split_data[i];
        auto const cut_value = d_matrix.GetFvalue(ridx, data.split_node.SplitIndex());
        if (isnan(cut_value)) {
          missing_bits.Set(ridx * num_candidates + i);
        } else {
          bool go_left;
          if (data.split_type == FeatureType::kCategorical) {
            go_left = common::Decision(data.node_cats.Bits(), cut_value);
          } else {
            go_left = cut_value <= data.split_node.SplitCond();
          }
          if (go_left) {
            decision_bits.Set(ridx * num_candidates + i);
          }
        }
      }
    });

    auto rc = collective::Success() << [&] {
      return collective::Allreduce(
          ctx_, linalg::MakeTensorView(ctx_, dh::ToSpan(decision_storage), decision_storage.size()),
          collective::Op::kBitwiseOR);
    } << [&] {
      return collective::Allreduce(
          ctx_, linalg::MakeTensorView(ctx_, dh::ToSpan(missing_storage), missing_storage.size()),
          collective::Op::kBitwiseAND);
    };
    collective::SafeColl(rc);

    partitioners_.front()->UpdatePositionBatch(
        nidx, left_nidx, right_nidx, split_data,
        [=] __device__(bst_uint ridx, int nidx_in_batch, NodeSplitData const& data) {
          auto const index = ridx * num_candidates + nidx_in_batch;
          bool go_left;
          if (missing_bits.Check(index)) {
            go_left = data.split_node.DefaultLeft();
          } else {
            go_left = decision_bits.Check(index);
          }
          return go_left;
        });
  }

  struct GoLeftOp {
    EllpackDeviceAccessor d_matrix;

    __device__ bool operator()(cuda_impl::RowIndexT ridx, NodeSplitData const& data) const {
      RegTree::Node const& node = data.split_node;
      // given a row index, returns the node id it belongs to
      float cut_value = d_matrix.GetFvalue(ridx, node.SplitIndex());
      // Missing value
      bool go_left = true;
      if (isnan(cut_value)) {
        go_left = node.DefaultLeft();
      } else {
        if (data.split_type == FeatureType::kCategorical) {
          go_left = common::Decision(data.node_cats.Bits(), cut_value);
        } else {
          go_left = cut_value <= node.SplitCond();
        }
      }
      return go_left;
    }
  };

  void UpdatePosition(DMatrix* p_fmat, std::vector<GPUExpandEntry> const& candidates,
                      RegTree* p_tree) {
    if (candidates.empty()) {
      return;
    }

    monitor.Start(__func__);

    std::vector<bst_node_t> nidx(candidates.size());
    std::vector<bst_node_t> left_nidx(candidates.size());
    std::vector<bst_node_t> right_nidx(candidates.size());
    std::vector<NodeSplitData> split_data(candidates.size());

    for (size_t i = 0; i < candidates.size(); i++) {
      auto const& e = candidates[i];
      RegTree::Node const& split_node = (*p_tree)[e.nid];
      auto split_type = p_tree->NodeSplitType(e.nid);
      nidx[i] = e.nid;
      left_nidx[i] = split_node.LeftChild();
      right_nidx[i] = split_node.RightChild();
      split_data[i] = NodeSplitData{split_node, split_type, evaluator_.GetDeviceNodeCats(e.nid)};

      CHECK_EQ(split_type == FeatureType::kCategorical, e.split.is_cat);
    }

    CHECK_EQ(p_fmat->NumBatches(), 1);
    for (auto const& page : p_fmat->GetBatches<EllpackPage>(ctx_, StaticBatch(true))) {
      auto d_matrix = page.Impl()->GetDeviceAccessor(ctx_->Device());

      if (p_fmat->Info().IsColumnSplit()) {
        UpdatePositionColumnSplit(d_matrix, split_data, nidx, left_nidx, right_nidx);
        monitor.Stop(__func__);
        return;
      }
      auto go_left = GoLeftOp{d_matrix};
      partitioners_.front()->UpdatePositionBatch(
          nidx, left_nidx, right_nidx, split_data,
          [=] __device__(cuda_impl::RowIndexT ridx, int /*nidx_in_batch*/,
                         const NodeSplitData& data) { return go_left(ridx, data); });
    }

    monitor.Stop(__func__);
  }

  // After tree update is finished, update the position of all training
  // instances to their final leaf. This information is used later to update the
  // prediction cache
  void FinalisePosition(DMatrix* p_fmat, RegTree const* p_tree, ObjInfo task, bst_idx_t n_samples,
                        HostDeviceVector<bst_node_t>* p_out_position) {
    if (!p_fmat->SingleColBlock() && task.UpdateTreeLeaf()) {
      LOG(FATAL) << "Current objective function can not be used with external memory.";
    }
    if (p_fmat->Info().num_row_ != n_samples) {
      // External memory with concatenation. Not supported.
      p_out_position->Resize(0);
      positions_.clear();
      return;
    }

    p_out_position->SetDevice(ctx_->Device());
    p_out_position->Resize(p_fmat->Info().num_row_);
    auto d_out_position = p_out_position->DeviceSpan();

    auto d_gpair = this->gpair;
    auto encode_op = [=] __device__(bst_idx_t ridx, bst_node_t nidx) {
      bool is_invalid = d_gpair[ridx].GetHess() - .0f == 0.f;
      return SamplePosition::Encode(nidx, !is_invalid);
    };  // NOLINT

    if (!p_fmat->SingleColBlock()) {
      for (std::size_t k = 0; k < partitioners_.size(); ++k) {
        auto& part = partitioners_.at(k);
        CHECK_EQ(part->GetNumNodes(), p_tree->NumNodes());
        auto base_ridx = batch_ptr_[k];
        auto n_samples = batch_ptr_.at(k + 1) - base_ridx;
        part->FinalisePosition(d_out_position.subspan(base_ridx, n_samples), base_ridx, encode_op);
      }
      dh::CopyTo(d_out_position, &positions_);
      return;
    }

    dh::caching_device_vector<uint32_t> categories;
    dh::CopyTo(p_tree->GetSplitCategories(), &categories);
    auto const& cat_segments = p_tree->GetSplitCategoriesPtr();
    auto d_categories = dh::ToSpan(categories);

    for (auto const& page : p_fmat->GetBatches<EllpackPage>(ctx_, StaticBatch(true))) {
      auto d_matrix = page.Impl()->GetDeviceAccessor(ctx_->Device());

      std::vector<NodeSplitData> split_data(p_tree->NumNodes());
      auto const& tree = *p_tree;
      for (std::size_t i = 0, n = split_data.size(); i < n; ++i) {
        RegTree::Node split_node = tree[i];
        auto split_type = p_tree->NodeSplitType(i);
        auto node_cats = common::GetNodeCats(d_categories, cat_segments[i]);
        split_data[i] = NodeSplitData{std::move(split_node), split_type, node_cats};
      }

      auto go_left_op = GoLeftOp{d_matrix};
      dh::caching_device_vector<NodeSplitData> d_split_data;
      dh::CopyTo(split_data, &d_split_data);
      auto s_split_data = dh::ToSpan(d_split_data);

      partitioners_.front()->FinalisePosition(
          d_out_position, page.BaseRowId(), [=] __device__(bst_idx_t row_id, bst_node_t nidx) {
            auto split_data = s_split_data[nidx];
            auto node = split_data.split_node;
            while (!node.IsLeaf()) {
              auto go_left = go_left_op(row_id, split_data);
              nidx = go_left ? node.LeftChild() : node.RightChild();
              node = s_split_data[nidx].split_node;
            }
            return encode_op(row_id, nidx);
          });
      dh::CopyTo(d_out_position, &positions_);
    }
  }

  bool UpdatePredictionCache(linalg::MatrixView<float> out_preds_d, RegTree const* p_tree) {
    if (positions_.empty()) {
      return false;
    }

    CHECK(p_tree);
    CHECK(out_preds_d.Device().IsCUDA());
    CHECK_EQ(out_preds_d.Device().ordinal, ctx_->Ordinal());

    auto d_position = dh::ToSpan(positions_);
    CHECK_EQ(out_preds_d.Size(), d_position.size());

    // Use the nodes from tree, the leaf value might be changed by the objective since the
    // last update tree call.
    dh::caching_device_vector<RegTree::Node> nodes;
    dh::CopyTo(p_tree->GetNodes(), &nodes);
    common::Span<RegTree::Node> d_nodes = dh::ToSpan(nodes);
    CHECK_EQ(out_preds_d.Shape(1), 1);
    dh::LaunchN(d_position.size(), ctx_->CUDACtx()->Stream(),
                [=] XGBOOST_DEVICE(std::size_t idx) mutable {
                  bst_node_t nidx = d_position[idx];
                  nidx = SamplePosition::Decode(nidx);
                  auto weight = d_nodes[nidx].LeafValue();
                  out_preds_d(idx, 0) += weight;
                });
    return true;
  }

  // num histograms is the number of contiguous histograms in memory to reduce over
  void AllReduceHist(MetaInfo const& info, bst_node_t nidx, int num_histograms) {
    monitor.Start(__func__);
    auto d_node_hist = hist.GetNodeHistogram(nidx);
    using ReduceT = typename std::remove_pointer<decltype(d_node_hist.data())>::type::ValueT;
    auto rc = collective::GlobalSum(
        ctx_, info,
        linalg::MakeVec(reinterpret_cast<ReduceT*>(d_node_hist.data()),
                        d_node_hist.size() * 2 * num_histograms, ctx_->Device()));
    SafeColl(rc);
    monitor.Stop(__func__);
  }

  /**
   * \brief Build GPU local histograms for the left and right child of some parent node
   */
  void BuildHistLeftRight(DMatrix* p_fmat, std::vector<GPUExpandEntry> const& candidates,
                          const RegTree& tree) {
    if (candidates.empty()) {
      return;
    }
    this->monitor.Start(__func__);
    // Some nodes we will manually compute histograms
    // others we will do by subtraction
    std::vector<int> hist_nidx;
    std::vector<int> subtraction_nidx;
    for (auto& e : candidates) {
      // Decide whether to build the left histogram or right histogram
      // Use sum of Hessian as a heuristic to select node with fewest training instances
      bool fewer_right = e.split.right_sum.GetQuantisedHess() < e.split.left_sum.GetQuantisedHess();
      if (fewer_right) {
        hist_nidx.emplace_back(tree[e.nid].RightChild());
        subtraction_nidx.emplace_back(tree[e.nid].LeftChild());
      } else {
        hist_nidx.emplace_back(tree[e.nid].LeftChild());
        subtraction_nidx.emplace_back(tree[e.nid].RightChild());
      }
    }
    std::vector<int> all_new = hist_nidx;
    all_new.insert(all_new.end(), subtraction_nidx.begin(), subtraction_nidx.end());
    // Allocate the histograms
    // Guaranteed contiguous memory
    hist.AllocateHistograms(ctx_, all_new);

    for (auto const& page : p_fmat->GetBatches<EllpackPage>(ctx_, StaticBatch(true))) {
      for (auto nidx : hist_nidx) {
        this->BuildHist(page.Impl(), nidx);
      }
    }

    // Reduce all in one go
    // This gives much better latency in a distributed setting
    // when processing a large batch
    this->AllReduceHist(p_fmat->Info(), hist_nidx.at(0), hist_nidx.size());

    for (size_t i = 0; i < subtraction_nidx.size(); i++) {
      auto build_hist_nidx = hist_nidx.at(i);
      auto subtraction_trick_nidx = subtraction_nidx.at(i);
      auto parent_nidx = candidates.at(i).nid;

      if (!this->SubtractionTrick(parent_nidx, build_hist_nidx, subtraction_trick_nidx)) {
        // Calculate other histogram manually
        for (auto const& page : p_fmat->GetBatches<EllpackPage>(ctx_, StaticBatch(true))) {
          this->BuildHist(page.Impl(), subtraction_trick_nidx);
        }
        this->AllReduceHist(p_fmat->Info(), subtraction_trick_nidx, 1);
      }
    }
    this->monitor.Stop(__func__);
  }

  void ApplySplit(const GPUExpandEntry& candidate, RegTree* p_tree) {
    RegTree& tree = *p_tree;

    // Sanity check - have we created a leaf with no training instances?
    if (!collective::IsDistributed() && partitioners_.size() == 1) {
      CHECK(partitioners_.front()->GetRows(candidate.nid).size() > 0)
          << "No training instances in this leaf!";
    }

    auto base_weight = candidate.base_weight;
    auto left_weight = candidate.left_weight * param.learning_rate;
    auto right_weight = candidate.right_weight * param.learning_rate;
    auto parent_hess =
        quantiser->ToFloatingPoint(candidate.split.left_sum + candidate.split.right_sum).GetHess();
    auto left_hess =
        quantiser->ToFloatingPoint(candidate.split.left_sum).GetHess();
    auto right_hess =
        quantiser->ToFloatingPoint(candidate.split.right_sum).GetHess();

    auto is_cat = candidate.split.is_cat;
    if (is_cat) {
      // should be set to nan in evaluation split.
      CHECK(common::CheckNAN(candidate.split.fvalue));
      std::vector<common::CatBitField::value_type> split_cats;

      auto h_cats = this->evaluator_.GetHostNodeCats(candidate.nid);
      auto n_bins_feature = cuts_->FeatureBins(candidate.split.findex);
      split_cats.resize(common::CatBitField::ComputeStorageSize(n_bins_feature), 0);
      CHECK_LE(split_cats.size(), h_cats.size());
      std::copy(h_cats.data(), h_cats.data() + split_cats.size(), split_cats.data());

      tree.ExpandCategorical(
          candidate.nid, candidate.split.findex, split_cats, candidate.split.dir == kLeftDir,
          base_weight, left_weight, right_weight, candidate.split.loss_chg, parent_hess,
          left_hess, right_hess);
    } else {
      CHECK(!common::CheckNAN(candidate.split.fvalue));
      tree.ExpandNode(candidate.nid, candidate.split.findex, candidate.split.fvalue,
                      candidate.split.dir == kLeftDir, base_weight, left_weight, right_weight,
                      candidate.split.loss_chg, parent_hess,
          left_hess, right_hess);
    }
    evaluator_.ApplyTreeSplit(candidate, p_tree);

    const auto& parent = tree[candidate.nid];
    interaction_constraints.Split(candidate.nid, parent.SplitIndex(), parent.LeftChild(),
                                  parent.RightChild());
  }

  GPUExpandEntry InitRoot(DMatrix* p_fmat, RegTree* p_tree) {
    constexpr bst_node_t kRootNIdx = 0;
    dh::XGBCachingDeviceAllocator<char> alloc;
    auto quantiser = *this->quantiser;
    auto gpair_it = dh::MakeTransformIterator<GradientPairInt64>(
        dh::tbegin(gpair),
        [=] __device__(auto const& gpair) { return quantiser.ToFixedPoint(gpair); });
    GradientPairInt64 root_sum_quantised =
        dh::Reduce(ctx_->CUDACtx()->CTP(), gpair_it, gpair_it + gpair.size(), GradientPairInt64{},
                   thrust::plus<GradientPairInt64>{});
    using ReduceT = typename decltype(root_sum_quantised)::ValueT;
    auto rc = collective::GlobalSum(
        ctx_, p_fmat->Info(), linalg::MakeVec(reinterpret_cast<ReduceT*>(&root_sum_quantised), 2));
    collective::SafeColl(rc);

    hist.AllocateHistograms(ctx_, {kRootNIdx});
    for (auto const& page : p_fmat->GetBatches<EllpackPage>(ctx_, StaticBatch(true))) {
      this->BuildHist(page.Impl(), kRootNIdx);
    }
    this->AllReduceHist(p_fmat->Info(), kRootNIdx, 1);

    // Remember root stats
    auto root_sum = quantiser.ToFloatingPoint(root_sum_quantised);
    p_tree->Stat(kRootNIdx).sum_hess = root_sum.GetHess();
    auto weight = CalcWeight(param, root_sum);
    p_tree->Stat(kRootNIdx).base_weight = weight;
    (*p_tree)[kRootNIdx].SetLeaf(param.learning_rate * weight);

    // Generate first split
    auto root_entry = this->EvaluateRootSplit(p_fmat, root_sum_quantised);
    return root_entry;
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair_all, DMatrix* p_fmat, ObjInfo const* task,
                  RegTree* p_tree, HostDeviceVector<bst_node_t>* p_out_position) {
    bool const is_single_block = p_fmat->SingleColBlock();
    bst_idx_t const n_samples = p_fmat->Info().num_row_;

    auto& tree = *p_tree;
    // Process maximum 32 nodes at a time
    Driver<GPUExpandEntry> driver(param, 32);

    monitor.Start("Reset");
    p_fmat = this->Reset(gpair_all, p_fmat);
    monitor.Stop("Reset");

    monitor.Start("InitRoot");
    driver.Push({this->InitRoot(p_fmat, p_tree)});
    monitor.Stop("InitRoot");

    // The set of leaves that can be expanded asynchronously
    auto expand_set = driver.Pop();
    while (!expand_set.empty()) {
      for (auto& candidate : expand_set) {
        this->ApplySplit(candidate, p_tree);
      }
      // Get the candidates we are allowed to expand further
      // e.g. We do not bother further processing nodes whose children are beyond max depth
      std::vector<GPUExpandEntry> filtered_expand_set;
      std::copy_if(expand_set.begin(), expand_set.end(), std::back_inserter(filtered_expand_set),
                   [&](const auto& e) { return driver.IsChildValid(e); });

      auto new_candidates =
          pinned.GetSpan<GPUExpandEntry>(filtered_expand_set.size() * 2, GPUExpandEntry{});
      // Update all the nodes if working with external memory, this saves us from working
      // with the finalize position call, which adds an additional iteration and requires
      // special handling for row index.
      this->UpdatePosition(p_fmat, is_single_block ? filtered_expand_set : expand_set, p_tree);

      this->BuildHistLeftRight(p_fmat, filtered_expand_set, tree);

      this->EvaluateSplits(p_fmat, filtered_expand_set, *p_tree, new_candidates);
      dh::DefaultStream().Sync();

      driver.Push(new_candidates.begin(), new_candidates.end());
      expand_set = driver.Pop();
    }
    // Row partitioner can have lesser nodes than the tree since we skip some leaf
    // nodes. These nodes are handled in the `FinalisePosition` call. However, a leaf can
    // be spliable before evaluation but invalid after evaluation as we have more
    // restrictions like min loss change after evalaution. Therefore, the check condition
    // is greater than or equal to.
    if (is_single_block) {
      CHECK_GE(p_tree->NumNodes(), this->partitioners_.front()->GetNumNodes());
    }
    this->FinalisePosition(p_fmat, p_tree, *task, n_samples, p_out_position);
  }
};

std::shared_ptr<common::HistogramCuts const> InitBatchCuts(Context const* ctx, DMatrix* p_fmat,
                                                           BatchParam batch,
                                                           std::vector<bst_idx_t>* p_batch_ptr) {
  std::vector<bst_idx_t>& batch_ptr = *p_batch_ptr;
  batch_ptr = {0};
  std::shared_ptr<common::HistogramCuts const> cuts;

  for (auto const& page : p_fmat->GetBatches<EllpackPage>(ctx, batch)) {
    batch_ptr.push_back(page.Size());
    cuts = page.Impl()->CutsShared();
    CHECK(cuts->cut_values_.DeviceCanRead());
  }
  CHECK(cuts);
  CHECK_EQ(p_fmat->NumBatches(), batch_ptr.size() - 1);
  std::partial_sum(batch_ptr.cbegin(), batch_ptr.cend(), batch_ptr.begin());
  return cuts;
}

class GPUHistMaker : public TreeUpdater {
  using GradientSumT = GradientPairPrecise;

 public:
  explicit GPUHistMaker(Context const* ctx, ObjInfo const* task) : TreeUpdater(ctx), task_{task} {};
  void Configure(const Args& args) override {
    // Used in test to count how many configurations are performed
    LOG(DEBUG) << "[GPU Hist]: Configure";
    hist_maker_param_.UpdateAllowUnknown(args);
    common::CheckComputeCapability();
    initialised_ = false;

    monitor_.Init("updater_gpu_hist");
  }

  void LoadConfig(Json const& in) override {
    auto const& config = get<Object const>(in);
    FromJson(config.at("hist_train_param"), &this->hist_maker_param_);
    initialised_ = false;
  }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["hist_train_param"] = ToJson(hist_maker_param_);
  }

  ~GPUHistMaker() override { dh::GlobalMemoryLogger().Log(); }

  void Update(TrainParam const* param, linalg::Matrix<GradientPair>* gpair, DMatrix* dmat,
              common::Span<HostDeviceVector<bst_node_t>> out_position,
              const std::vector<RegTree*>& trees) override {
    monitor_.Start(__func__);

    CHECK_EQ(gpair->Shape(1), 1) << MTNotImplemented();
    auto gpair_hdv = gpair->Data();
    // build tree
    std::size_t t_idx{0};
    for (xgboost::RegTree* tree : trees) {
      this->UpdateTree(param, gpair_hdv, dmat, tree, &out_position[t_idx]);
      this->hist_maker_param_.CheckTreesSynchronized(ctx_, tree);
      ++t_idx;
    }
    dh::safe_cuda(hipGetLastError());
    monitor_.Stop(__func__);
  }

  void InitDataOnce(TrainParam const* param, DMatrix* p_fmat) {
    CHECK_GE(ctx_->Ordinal(), 0) << "Must have at least one device";

    // Synchronise the column sampling seed
    std::uint32_t column_sampling_seed = common::GlobalRandom()();
    SafeColl(collective::Broadcast(
        ctx_, linalg::MakeVec(&column_sampling_seed, sizeof(column_sampling_seed)), 0));
    this->column_sampler_ = std::make_shared<common::ColumnSampler>(column_sampling_seed);

    dh::safe_cuda(hipSetDevice(ctx_->Ordinal()));
    p_fmat->Info().feature_types.SetDevice(ctx_->Device());

    std::vector<bst_idx_t> batch_ptr;
    auto batch = HistBatch(*param);
    auto cuts = InitBatchCuts(ctx_, p_fmat, batch, &batch_ptr);

    this->maker = std::make_unique<GPUHistMakerDevice>(ctx_, *param, column_sampler_, batch,
                                                       p_fmat->Info(), batch_ptr, cuts);

    p_last_fmat_ = p_fmat;
    initialised_ = true;
  }

  void InitData(TrainParam const* param, DMatrix* dmat, RegTree const* p_tree) {
    if (!initialised_) {
      monitor_.Start("InitDataOnce");
      this->InitDataOnce(param, dmat);
      monitor_.Stop("InitDataOnce");
    }
    p_last_tree_ = p_tree;
    CHECK(hist_maker_param_.GetInitialised());
  }

  void UpdateTree(TrainParam const* param, HostDeviceVector<GradientPair>* gpair, DMatrix* p_fmat,
                  RegTree* p_tree, HostDeviceVector<bst_node_t>* p_out_position) {
    monitor_.Start("InitData");
    this->InitData(param, p_fmat, p_tree);
    monitor_.Stop("InitData");

    gpair->SetDevice(ctx_->Device());
    maker->UpdateTree(gpair, p_fmat, task_, p_tree, p_out_position);
  }

  bool UpdatePredictionCache(const DMatrix* data, linalg::MatrixView<float> p_out_preds) override {
    if (maker == nullptr || p_last_fmat_ == nullptr || p_last_fmat_ != data) {
      return false;
    }
    monitor_.Start(__func__);
    bool result = maker->UpdatePredictionCache(p_out_preds, p_last_tree_);
    monitor_.Stop(__func__);
    return result;
  }

  std::unique_ptr<GPUHistMakerDevice> maker;  // NOLINT

  [[nodiscard]] char const* Name() const override { return "grow_gpu_hist"; }
  [[nodiscard]] bool HasNodePosition() const override { return true; }

 private:
  bool initialised_{false};

  HistMakerTrainParam hist_maker_param_;

  DMatrix* p_last_fmat_{nullptr};
  RegTree const* p_last_tree_{nullptr};
  ObjInfo const* task_{nullptr};

  common::Monitor monitor_;
  std::shared_ptr<common::ColumnSampler> column_sampler_;
};

XGBOOST_REGISTER_TREE_UPDATER(GPUHistMaker, "grow_gpu_hist")
    .describe("Grow tree with GPU.")
    .set_body([](Context const* ctx, ObjInfo const* task) {
      return new GPUHistMaker(ctx, task);
    });

class GPUGlobalApproxMaker : public TreeUpdater {
 public:
  explicit GPUGlobalApproxMaker(Context const* ctx, ObjInfo const* task)
      : TreeUpdater(ctx), task_{task} {};
  void Configure(Args const& args) override {
    // Used in test to count how many configurations are performed
    LOG(DEBUG) << "[GPU Approx]: Configure";
    hist_maker_param_.UpdateAllowUnknown(args);
    if (hist_maker_param_.max_cached_hist_node != HistMakerTrainParam::DefaultNodes()) {
      LOG(WARNING) << "The `max_cached_hist_node` is ignored in GPU.";
    }
    common::CheckComputeCapability();
    initialised_ = false;

    monitor_.Init(this->Name());
  }

  void LoadConfig(Json const& in) override {
    auto const& config = get<Object const>(in);
    FromJson(config.at("hist_train_param"), &this->hist_maker_param_);
    initialised_ = false;
  }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["hist_train_param"] = ToJson(hist_maker_param_);
  }
  ~GPUGlobalApproxMaker() override { dh::GlobalMemoryLogger().Log(); }

  void Update(TrainParam const* param, linalg::Matrix<GradientPair>* gpair, DMatrix* p_fmat,
              common::Span<HostDeviceVector<bst_node_t>> out_position,
              const std::vector<RegTree*>& trees) override {
    monitor_.Start(__func__);

    this->InitDataOnce(p_fmat);
    // build tree
    hess_.resize(gpair->Size());
    auto hess = dh::ToSpan(hess_);

    gpair->SetDevice(ctx_->Device());
    auto d_gpair = gpair->Data()->ConstDeviceSpan();
    auto cuctx = ctx_->CUDACtx();
    thrust::transform(cuctx->CTP(), dh::tcbegin(d_gpair), dh::tcend(d_gpair), dh::tbegin(hess),
                      [=] XGBOOST_DEVICE(GradientPair const& g) { return g.GetHess(); });

    auto const& info = p_fmat->Info();
    info.feature_types.SetDevice(ctx_->Device());

    std::vector<bst_idx_t> batch_ptr;
    auto batch = ApproxBatch(*param, hess, *task_);
    auto cuts = InitBatchCuts(ctx_, p_fmat, batch, &batch_ptr);
    batch.regen = false;  // Regen only at the beginning of the iteration.

    this->maker_ = std::make_unique<GPUHistMakerDevice>(ctx_, *param, column_sampler_, batch,
                                                        p_fmat->Info(), batch_ptr, cuts);

    std::size_t t_idx{0};
    for (xgboost::RegTree* tree : trees) {
      this->UpdateTree(gpair->Data(), p_fmat, tree, &out_position[t_idx]);
      this->hist_maker_param_.CheckTreesSynchronized(ctx_, tree);
      ++t_idx;
    }

    monitor_.Stop(__func__);
  }

  void InitDataOnce(DMatrix* p_fmat) {
    if (this->initialised_) {
      return;
    }

    monitor_.Start(__func__);
    CHECK(ctx_->IsCUDA()) << error::InvalidCUDAOrdinal();
    uint32_t column_sampling_seed = common::GlobalRandom()();
    this->column_sampler_ = std::make_shared<common::ColumnSampler>(column_sampling_seed);

    p_last_fmat_ = p_fmat;
    initialised_ = true;
    monitor_.Stop(__func__);
  }

  void InitData(DMatrix* p_fmat, RegTree const* p_tree) {
    this->InitDataOnce(p_fmat);
    p_last_tree_ = p_tree;
    CHECK(hist_maker_param_.GetInitialised());
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* p_fmat, RegTree* p_tree,
                  HostDeviceVector<bst_node_t>* p_out_position) {
    monitor_.Start("InitData");
    this->InitData(p_fmat, p_tree);
    monitor_.Stop("InitData");

    gpair->SetDevice(ctx_->Device());
    maker_->UpdateTree(gpair, p_fmat, task_, p_tree, p_out_position);
  }

  bool UpdatePredictionCache(const DMatrix* data, linalg::MatrixView<float> p_out_preds) override {
    if (maker_ == nullptr || p_last_fmat_ == nullptr || p_last_fmat_ != data) {
      return false;
    }
    monitor_.Start(__func__);
    bool result = maker_->UpdatePredictionCache(p_out_preds, p_last_tree_);
    monitor_.Stop(__func__);
    return result;
  }

  [[nodiscard]] char const* Name() const override { return "grow_gpu_approx"; }
  [[nodiscard]] bool HasNodePosition() const override { return true; }

 private:
  bool initialised_{false};

  HistMakerTrainParam hist_maker_param_;
  dh::device_vector<float> hess_;
  std::shared_ptr<common::ColumnSampler> column_sampler_;
  std::unique_ptr<GPUHistMakerDevice> maker_;

  DMatrix* p_last_fmat_{nullptr};
  RegTree const* p_last_tree_{nullptr};
  ObjInfo const* task_{nullptr};

  common::Monitor monitor_;
};

XGBOOST_REGISTER_TREE_UPDATER(GPUApproxMaker, "grow_gpu_approx")
    .describe("Grow tree with GPU.")
    .set_body([](Context const* ctx, ObjInfo const* task) {
      return new GPUGlobalApproxMaker(ctx, task);
    });
}  // namespace xgboost::tree
