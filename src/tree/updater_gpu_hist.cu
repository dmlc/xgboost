#include "hip/hip_runtime.h"
/*!
 * Copyright 2017 XGBoost contributors
 */
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <xgboost/tree_updater.h>
#include <algorithm>
#include <cmath>
#include <memory>
#include <queue>
#include <utility>
#include <vector>
#include "../common/compressed_iterator.h"
#include "../common/device_helpers.cuh"
#include "../common/hist_util.h"
#include "../common/host_device_vector.h"
#include "../common/timer.h"
#include "param.h"
#include "updater_gpu_common.cuh"

namespace xgboost {
namespace tree {

DMLC_REGISTRY_FILE_TAG(updater_gpu_hist);

using GradientPairSumT = GradientPairPrecise;

template <int BLOCK_THREADS, typename ReduceT, typename TempStorageT>
__device__ GradientPairSumT ReduceFeature(const GradientPairSumT* begin,
                                     const GradientPairSumT* end,
                                     TempStorageT* temp_storage) {
  __shared__ hipcub::Uninitialized<GradientPairSumT> uninitialized_sum;
  GradientPairSumT& shared_sum = uninitialized_sum.Alias();

  GradientPairSumT local_sum = GradientPairSumT();
  for (auto itr = begin; itr < end; itr += BLOCK_THREADS) {
    bool thread_active = itr + threadIdx.x < end;
    // Scan histogram
    GradientPairSumT bin = thread_active ? *(itr + threadIdx.x) : GradientPairSumT();
    local_sum += bin;
  }
  local_sum = ReduceT(temp_storage->sum_reduce).Reduce(local_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    shared_sum = local_sum;
  }
  __syncthreads();

  return shared_sum;
}

template <int BLOCK_THREADS, typename ReduceT, typename scan_t,
          typename max_ReduceT, typename TempStorageT>
__device__ void EvaluateFeature(int fidx, const GradientPairSumT* hist,
                                const int* feature_segments, float min_fvalue,
                                const float* gidx_fvalue_map,
                                DeviceSplitCandidate* best_split,
                                const DeviceNodeStats& node,
                                const GPUTrainingParam& param,
                                TempStorageT* temp_storage, int constraint,
                                const ValueConstraint& value_constraint) {
  int gidx_begin = feature_segments[fidx];
  int gidx_end = feature_segments[fidx + 1];

  GradientPairSumT feature_sum = ReduceFeature<BLOCK_THREADS, ReduceT>(
      hist + gidx_begin, hist + gidx_end, temp_storage);

  auto prefix_op = SumCallbackOp<GradientPairSumT>();
  for (int scan_begin = gidx_begin; scan_begin < gidx_end;
       scan_begin += BLOCK_THREADS) {
    bool thread_active = scan_begin + threadIdx.x < gidx_end;

    GradientPairSumT bin =
        thread_active ? hist[scan_begin + threadIdx.x] : GradientPairSumT();
    scan_t(temp_storage->scan).ExclusiveScan(bin, bin, hipcub::Sum(), prefix_op);

    // Calculate  gain
    GradientPairSumT parent_sum = GradientPairSumT(node.sum_gradients);

    GradientPairSumT missing = parent_sum - feature_sum;

    bool missing_left = true;
    const float null_gain = -FLT_MAX;
    float gain = null_gain;
    if (thread_active) {
      gain = LossChangeMissing(bin, missing, parent_sum, node.root_gain, param,
                              constraint, value_constraint, missing_left);
    }

    __syncthreads();

    // Find thread with best gain
    hipcub::KeyValuePair<int, float> tuple(threadIdx.x, gain);
    hipcub::KeyValuePair<int, float> best =
        max_ReduceT(temp_storage->max_reduce).Reduce(tuple, hipcub::ArgMax());

    __shared__ hipcub::KeyValuePair<int, float> block_max;
    if (threadIdx.x == 0) {
      block_max = best;
    }

    __syncthreads();

    // Best thread updates split
    if (threadIdx.x == block_max.key) {
      int gidx = scan_begin + threadIdx.x;
      float fvalue =
          gidx == gidx_begin ? min_fvalue : gidx_fvalue_map[gidx - 1];

      GradientPairSumT left = missing_left ? bin + missing : bin;
      GradientPairSumT right = parent_sum - left;

      best_split->Update(gain, missing_left ? kLeftDir : kRightDir, fvalue, fidx,
                         GradientPair(left), GradientPair(right), param);
    }
    __syncthreads();
  }
}

template <int BLOCK_THREADS>
__global__ void evaluate_split_kernel(
    const GradientPairSumT* d_hist, int nidx, uint64_t n_features,
    DeviceNodeStats nodes, const int* d_feature_segments,
    const float* d_fidx_min_map, const float* d_gidx_fvalue_map,
    GPUTrainingParam gpu_param, DeviceSplitCandidate* d_split,
    ValueConstraint value_constraint, int* d_monotonic_constraints) {
  typedef hipcub::KeyValuePair<int, float> ArgMaxT;
  typedef hipcub::BlockScan<GradientPairSumT, BLOCK_THREADS, hipcub::BLOCK_SCAN_WARP_SCANS>
      BlockScanT;
  typedef hipcub::BlockReduce<ArgMaxT, BLOCK_THREADS> MaxReduceT;

  typedef hipcub::BlockReduce<GradientPairSumT, BLOCK_THREADS> SumReduceT;

  union TempStorage {
    typename BlockScanT::TempStorage scan;
    typename MaxReduceT::TempStorage max_reduce;
    typename SumReduceT::TempStorage sum_reduce;
  };

  __shared__ hipcub::Uninitialized<DeviceSplitCandidate> uninitialized_split;
  DeviceSplitCandidate& best_split = uninitialized_split.Alias();
  __shared__ TempStorage temp_storage;

  if (threadIdx.x == 0) {
    best_split = DeviceSplitCandidate();
  }

  __syncthreads();

  auto fidx = blockIdx.x;
  auto constraint = d_monotonic_constraints[fidx];
  EvaluateFeature<BLOCK_THREADS, SumReduceT, BlockScanT, MaxReduceT>(
      fidx, d_hist, d_feature_segments, d_fidx_min_map[fidx], d_gidx_fvalue_map,
      &best_split, nodes, gpu_param, &temp_storage, constraint,
      value_constraint);

  __syncthreads();

  if (threadIdx.x == 0) {
    // Record best loss
    d_split[fidx] = best_split;
  }
}

// Find a gidx value for a given feature otherwise return -1 if not found
template <typename GidxIterT>
__device__ int BinarySearchRow(bst_uint begin, bst_uint end, GidxIterT data,
                               int fidx_begin, int fidx_end) {
  bst_uint previous_middle = UINT32_MAX;
  while (end != begin) {
    auto middle = begin + (end - begin) / 2;
    if (middle == previous_middle) {
      break;
    }
    previous_middle = middle;

    auto gidx = data[middle];

    if (gidx >= fidx_begin && gidx < fidx_end) {
      return gidx;
    } else if (gidx < fidx_begin) {
      begin = middle;
    } else {
      end = middle;
    }
  }
  // Value is missing
  return -1;
}

/**
 * \struct  DeviceHistogram
 *
 * \summary Data storage for node histograms on device. Automatically expands.
 *
 * \author  Rory
 * \date    28/07/2018
 */

struct DeviceHistogram {
  std::map<int, size_t>
      nidx_map;  // Map nidx to starting index of its histogram
  thrust::device_vector<GradientPairSumT> data;
  int n_bins;
  int device_idx;
  void Init(int device_idx, int n_bins) {
    this->n_bins = n_bins;
    this->device_idx = device_idx;
  }

  void Reset() {
    dh::safe_cuda(hipSetDevice(device_idx));
    thrust::fill(data.begin(), data.end(), GradientPairSumT());
  }

  /**
   * \summary   Return pointer to histogram memory for a given node. Be aware that this function
   *            may reallocate the underlying memory, invalidating previous pointers.
   *
   * \author    Rory
   * \date  28/07/2018
   *
   * \param nidx    Tree node index.
   *
   * \return    hist pointer.
   */

  GradientPairSumT* GetHistPtr(int nidx) {
    if (nidx_map.find(nidx) == nidx_map.end()) {
      // Append new node histogram
      nidx_map[nidx] = data.size();
      dh::safe_cuda(hipSetDevice(device_idx));
      data.resize(data.size() + n_bins, GradientPairSumT());
    }
    return data.data().get() + nidx_map[nidx];
  }
};

struct CalcWeightTrainParam {
  float min_child_weight;
  float reg_alpha;
  float reg_lambda;
  float max_delta_step;
  float learning_rate;
  XGBOOST_DEVICE explicit CalcWeightTrainParam(const TrainParam& p)
      : min_child_weight(p.min_child_weight),
        reg_alpha(p.reg_alpha),
        reg_lambda(p.reg_lambda),
        max_delta_step(p.max_delta_step),
        learning_rate(p.learning_rate) {}
};

__global__ void compress_bin_ellpack_k
(common::CompressedBufferWriter wr, common::CompressedByteT* __restrict__ buffer,
 const size_t* __restrict__ row_ptrs,
 const Entry* __restrict__ entries,
 const float* __restrict__ cuts, const size_t* __restrict__ cut_rows,
 size_t base_row, size_t n_rows, size_t row_ptr_begin, size_t row_stride,
 unsigned int null_gidx_value) {
  size_t irow = threadIdx.x + size_t(blockIdx.x) * blockDim.x;
  int ifeature = threadIdx.y + blockIdx.y * blockDim.y;
  if (irow >= n_rows || ifeature >= row_stride)
    return;
  int row_size = static_cast<int>(row_ptrs[irow + 1] - row_ptrs[irow]);
  unsigned int bin = null_gidx_value;
  if (ifeature < row_size) {
    Entry entry = entries[row_ptrs[irow] - row_ptr_begin + ifeature];
    int feature = entry.index;
    float fvalue = entry.fvalue;
    const float *feature_cuts = &cuts[cut_rows[feature]];
    int ncuts = cut_rows[feature + 1] - cut_rows[feature];
    bin = dh::UpperBound(feature_cuts, ncuts, fvalue);
    if (bin >= ncuts)
      bin = ncuts - 1;
    bin += cut_rows[feature];
  }
  wr.AtomicWriteSymbol(buffer, bin, (irow + base_row) * row_stride + ifeature);
}

__global__ void sharedMemHistKernel(size_t row_stride,
                                    const bst_uint* d_ridx,
                                    common::CompressedIterator<uint32_t> d_gidx,
                                    int null_gidx_value,
                                    GradientPairSumT* d_node_hist,
                                    const GradientPair* d_gpair,
                                    size_t segment_begin,
                                    size_t n_elements) {
  extern __shared__ char smem[];
  GradientPairSumT* smem_arr = reinterpret_cast<GradientPairSumT*>(smem); // NOLINT
  for (auto i : dh::BlockStrideRange(0, null_gidx_value)) {
    smem_arr[i] = GradientPairSumT();
  }
  __syncthreads();
  for (auto idx : dh::GridStrideRange(static_cast<size_t>(0), n_elements)) {
    int ridx = d_ridx[idx / row_stride + segment_begin];
    int gidx = d_gidx[ridx * row_stride + idx % row_stride];
    if (gidx != null_gidx_value) {
      AtomicAddGpair(smem_arr + gidx, d_gpair[ridx]);
    }
  }
  __syncthreads();
  for (auto i : dh::BlockStrideRange(0, null_gidx_value)) {
    AtomicAddGpair(d_node_hist + i, smem_arr[i]);
  }
}

// Manage memory for a single GPU
struct DeviceShard {
  struct Segment {
    size_t begin;
    size_t end;

    Segment() : begin(0), end(0) {}

    Segment(size_t begin, size_t end) : begin(begin), end(end) {
      CHECK_GE(end, begin);
    }
    size_t Size() const { return end - begin; }
  };

  int device_idx;
  int normalised_device_idx;  // Device index counting from param.gpu_id
  dh::BulkAllocator<dh::MemoryType::kDevice> ba;
  dh::DVec<common::CompressedByteT> gidx_buffer;
  dh::DVec<GradientPair> gpair;
  dh::DVec2<bst_uint> ridx;  // Row index relative to this shard
  dh::DVec2<int> position;
  std::vector<Segment> ridx_segments;
  dh::DVec<int> feature_segments;
  dh::DVec<float> gidx_fvalue_map;
  dh::DVec<float> min_fvalue;
  dh::DVec<int> monotone_constraints;
  dh::DVec<bst_float> prediction_cache;
  std::vector<GradientPair> node_sum_gradients;
  dh::DVec<GradientPair> node_sum_gradients_d;
  thrust::device_vector<size_t> row_ptrs;
  common::CompressedIterator<uint32_t> gidx;
  size_t row_stride;
  bst_uint row_begin_idx;  // The row offset for this shard
  bst_uint row_end_idx;
  bst_uint n_rows;
  int n_bins;
  int null_gidx_value;
  DeviceHistogram hist;
  TrainParam param;
  bool prediction_cache_initialised;
  bool can_use_smem_atomics;

  int64_t* tmp_pinned;  // Small amount of staging memory

  std::vector<hipStream_t> streams;

  dh::CubMemory temp_memory;

  // TODO(canonizer): do add support multi-batch DMatrix here
  DeviceShard(int device_idx, int normalised_device_idx,
              bst_uint row_begin, bst_uint row_end, TrainParam param)
    : device_idx(device_idx),
      normalised_device_idx(normalised_device_idx),
      row_begin_idx(row_begin),
      row_end_idx(row_end),
      row_stride(0),
      n_rows(row_end - row_begin),
      n_bins(0),
      null_gidx_value(0),
      param(param),
      prediction_cache_initialised(false),
      can_use_smem_atomics(false) {}

  void InitRowPtrs(const SparsePage& row_batch) {
    dh::safe_cuda(hipSetDevice(device_idx));
    row_ptrs.resize(n_rows + 1);
    thrust::copy(row_batch.offset.data() + row_begin_idx,
                 row_batch.offset.data() + row_end_idx + 1,
                 row_ptrs.begin());
    auto row_iter = row_ptrs.begin();
    auto get_size = [=] __device__(size_t row) {
      return row_iter[row + 1] - row_iter[row];
    }; // NOLINT

    auto counting = thrust::make_counting_iterator(size_t(0));
    using TransformT = thrust::transform_iterator<decltype(get_size),
      decltype(counting), size_t>;
    TransformT row_size_iter = TransformT(counting, get_size);
    row_stride = thrust::reduce(row_size_iter, row_size_iter + n_rows, 0,
                                thrust::maximum<size_t>());
  }

  void InitCompressedData(const common::HistCutMatrix& hmat, const SparsePage& row_batch) {
    n_bins = hmat.row_ptr.back();
    null_gidx_value = hmat.row_ptr.back();

    // copy cuts to the GPU
    dh::safe_cuda(hipSetDevice(device_idx));
    thrust::device_vector<float> cuts_d(hmat.cut);
    thrust::device_vector<size_t> cut_row_ptrs_d(hmat.row_ptr);

    // allocate compressed bin data
    int num_symbols = n_bins + 1;
    size_t compressed_size_bytes =
        common::CompressedBufferWriter::CalculateBufferSize(row_stride * n_rows,
                                                            num_symbols);

    CHECK(!(param.max_leaves == 0 && param.max_depth == 0))
        << "Max leaves and max depth cannot both be unconstrained for "
           "gpu_hist.";
    ba.Allocate(device_idx, param.silent, &gidx_buffer, compressed_size_bytes);
    gidx_buffer.Fill(0);

    int nbits = common::detail::SymbolBits(num_symbols);

    // bin and compress entries in batches of rows
    size_t gpu_batch_nrows = std::min
      (dh::TotalMemory(device_idx) / (16 * row_stride * sizeof(Entry)),
       static_cast<size_t>(n_rows));

    thrust::device_vector<Entry> entries_d(gpu_batch_nrows * row_stride);

    size_t gpu_nbatches = dh::DivRoundUp(n_rows, gpu_batch_nrows);
    for (size_t gpu_batch = 0; gpu_batch < gpu_nbatches; ++gpu_batch) {
      size_t batch_row_begin = gpu_batch * gpu_batch_nrows;
      size_t batch_row_end = (gpu_batch + 1) * gpu_batch_nrows;
      if (batch_row_end > n_rows) {
        batch_row_end = n_rows;
      }
      size_t batch_nrows = batch_row_end - batch_row_begin;
      size_t n_entries =
        row_batch.offset[row_begin_idx + batch_row_end] -
        row_batch.offset[row_begin_idx + batch_row_begin];
      dh::safe_cuda
        (hipMemcpy
         (entries_d.data().get(),
          &row_batch.data[row_batch.offset[row_begin_idx + batch_row_begin]],
          n_entries * sizeof(Entry), hipMemcpyDefault));
      dim3 block3(32, 8, 1);
      dim3 grid3(dh::DivRoundUp(n_rows, block3.x),
                 dh::DivRoundUp(row_stride, block3.y), 1);
      compress_bin_ellpack_k<<<grid3, block3>>>
        (common::CompressedBufferWriter(num_symbols), gidx_buffer.Data(),
         row_ptrs.data().get() + batch_row_begin,
         entries_d.data().get(), cuts_d.data().get(), cut_row_ptrs_d.data().get(),
         batch_row_begin, batch_nrows,
         row_batch.offset[row_begin_idx + batch_row_begin],
         row_stride, null_gidx_value);

      dh::safe_cuda(hipGetLastError());
      dh::safe_cuda(hipDeviceSynchronize());
    }

    // free the memory that is no longer needed
    row_ptrs.resize(0);
    row_ptrs.shrink_to_fit();
    entries_d.resize(0);
    entries_d.shrink_to_fit();

    gidx = common::CompressedIterator<uint32_t>(gidx_buffer.Data(), num_symbols);

    // allocate the rest
    int max_nodes =
        param.max_leaves > 0 ? param.max_leaves * 2 : MaxNodesDepth(param.max_depth);
    ba.Allocate(device_idx, param.silent,
                &gpair, n_rows, &ridx, n_rows, &position, n_rows,
                &prediction_cache, n_rows, &node_sum_gradients_d, max_nodes,
                &feature_segments, hmat.row_ptr.size(), &gidx_fvalue_map,
                hmat.cut.size(), &min_fvalue, hmat.min_val.size(),
                &monotone_constraints, param.monotone_constraints.size());
    gidx_fvalue_map = hmat.cut;
    min_fvalue = hmat.min_val;
    feature_segments = hmat.row_ptr;
    monotone_constraints = param.monotone_constraints;

    node_sum_gradients.resize(max_nodes);
    ridx_segments.resize(max_nodes);

    // check if we can use shared memory for building histograms
    // (assuming atleast we need 2 CTAs per SM to maintain decent latency hiding)
    auto histogram_size = sizeof(GradientPairSumT) * null_gidx_value;
    auto max_smem = dh::MaxSharedMemory(device_idx);
    can_use_smem_atomics = histogram_size <= max_smem;

    // Init histogram
    hist.Init(device_idx, hmat.row_ptr.back());

    dh::safe_cuda(hipHostMalloc(&tmp_pinned, sizeof(int64_t)));
  }

  ~DeviceShard() {
    for (auto& stream : streams) {
      dh::safe_cuda(hipStreamDestroy(stream));
    }
    dh::safe_cuda(hipHostFree(tmp_pinned));
  }

  // Get vector of at least n initialised streams
  std::vector<hipStream_t>& GetStreams(int n) {
    if (n > streams.size()) {
      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamDestroy(stream));
      }

      streams.clear();
      streams.resize(n);

      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamCreate(&stream));
      }
    }

    return streams;
  }

  // Reset values for each update iteration
  void Reset(HostDeviceVector<GradientPair>* dh_gpair) {
    dh::safe_cuda(hipSetDevice(device_idx));
    position.CurrentDVec().Fill(0);
    std::fill(node_sum_gradients.begin(), node_sum_gradients.end(),
              GradientPair());

    thrust::sequence(ridx.CurrentDVec().tbegin(), ridx.CurrentDVec().tend());

    std::fill(ridx_segments.begin(), ridx_segments.end(), Segment(0, 0));
    ridx_segments.front() = Segment(0, ridx.Size());
    this->gpair.copy(dh_gpair->tbegin(device_idx), dh_gpair->tend(device_idx));
    SubsampleGradientPair(&gpair, param.subsample, row_begin_idx);
    hist.Reset();
  }

  void BuildHistUsingGlobalMem(int nidx) {
    auto segment = ridx_segments[nidx];
    auto d_node_hist = hist.GetHistPtr(nidx);
    auto d_gidx = gidx;
    auto d_ridx = ridx.Current();
    auto d_gpair = gpair.Data();
    auto row_stride = this->row_stride;
    auto null_gidx_value = this->null_gidx_value;
    auto n_elements = segment.Size() * row_stride;

    dh::LaunchN(device_idx, n_elements, [=] __device__(size_t idx) {
      int ridx = d_ridx[(idx / row_stride) + segment.begin];
      int gidx = d_gidx[ridx * row_stride + idx % row_stride];

      if (gidx != null_gidx_value) {
        AtomicAddGpair(d_node_hist + gidx, d_gpair[ridx]);
      }
    });
  }

  void BuildHistUsingSharedMem(int nidx) {
    auto segment = ridx_segments[nidx];
    auto segment_begin = segment.begin;
    auto d_node_hist = hist.GetHistPtr(nidx);
    auto d_gidx = gidx;
    auto d_ridx = ridx.Current();
    auto d_gpair = gpair.Data();
    auto row_stride = this->row_stride;
    auto null_gidx_value = this->null_gidx_value;
    auto n_elements = segment.Size() * row_stride;

    const size_t smem_size = sizeof(GradientPairSumT) * null_gidx_value;
    const int items_per_thread = 8;
    const int block_threads = 256;
    const int grid_size =
        static_cast<int>(dh::DivRoundUp(n_elements,
                                        items_per_thread * block_threads));
    if (grid_size <= 0) {
      return;
    }
    dh::safe_cuda(hipSetDevice(device_idx));
    sharedMemHistKernel<<<grid_size, block_threads, smem_size>>>
        (row_stride, d_ridx, d_gidx, null_gidx_value, d_node_hist, d_gpair,
         segment_begin, n_elements);
  }

  void BuildHist(int nidx) {
    if (can_use_smem_atomics) {
      BuildHistUsingSharedMem(nidx);
    } else {
      BuildHistUsingGlobalMem(nidx);
    }
  }

  void SubtractionTrick(int nidx_parent, int nidx_histogram,
                        int nidx_subtraction) {
    // Make sure histograms are already allocated
    hist.GetHistPtr(nidx_parent);
    hist.GetHistPtr(nidx_histogram);
    hist.GetHistPtr(nidx_subtraction);
    auto d_node_hist_parent = hist.GetHistPtr(nidx_parent);
    auto d_node_hist_histogram = hist.GetHistPtr(nidx_histogram);
    auto d_node_hist_subtraction = hist.GetHistPtr(nidx_subtraction);

    dh::LaunchN(device_idx, hist.n_bins, [=] __device__(size_t idx) {
      d_node_hist_subtraction[idx] =
          d_node_hist_parent[idx] - d_node_hist_histogram[idx];
    });
  }

  __device__ void CountLeft(int64_t* d_count, int val, int left_nidx) {
    unsigned ballot = __ballot(val == left_nidx);
    if (threadIdx.x % 32 == 0) {
      atomicAdd(reinterpret_cast<unsigned long long*>(d_count),    // NOLINT
                static_cast<unsigned long long>(__popc(ballot)));  // NOLINT
    }
  }

  void UpdatePosition(int nidx, int left_nidx, int right_nidx, int fidx,
                      int split_gidx, bool default_dir_left, bool is_dense,
                      int fidx_begin, int fidx_end) {
    dh::safe_cuda(hipSetDevice(device_idx));
    temp_memory.LazyAllocate(sizeof(int64_t));
    auto d_left_count = temp_memory.Pointer<int64_t>();
    dh::safe_cuda(hipMemset(d_left_count, 0, sizeof(int64_t)));
    auto segment = ridx_segments[nidx];
    auto d_ridx = ridx.Current();
    auto d_position = position.Current();
    auto d_gidx = gidx;
    auto row_stride = this->row_stride;
    dh::LaunchN<1, 512>(
        device_idx, segment.Size(), [=] __device__(bst_uint idx) {
          idx += segment.begin;
          auto ridx = d_ridx[idx];
          auto row_begin = row_stride * ridx;
          auto row_end = row_begin + row_stride;
          auto gidx = -1;
          if (is_dense) {
            gidx = d_gidx[row_begin + fidx];
          } else {
            gidx = BinarySearchRow(row_begin, row_end, d_gidx, fidx_begin,
                                   fidx_end);
          }

          int position;
          if (gidx >= 0) {
            // Feature is found
            position = gidx <= split_gidx ? left_nidx : right_nidx;
          } else {
            // Feature is missing
            position = default_dir_left ? left_nidx : right_nidx;
          }

          CountLeft(d_left_count, position, left_nidx);
          d_position[idx] = position;
        });

    dh::safe_cuda(hipMemcpy(tmp_pinned, d_left_count, sizeof(int64_t),
                             hipMemcpyDeviceToHost));
    auto left_count = *tmp_pinned;

    SortPosition(segment, left_nidx, right_nidx);
    // dh::safe_cuda(hipStreamSynchronize(stream));
    ridx_segments[left_nidx] =
        Segment(segment.begin, segment.begin + left_count);
    ridx_segments[right_nidx] =
        Segment(segment.begin + left_count, segment.end);
  }

  void SortPosition(const Segment& segment, int left_nidx, int right_nidx) {
    int min_bits = 0;
    int max_bits = static_cast<int>(
        std::ceil(std::log2((std::max)(left_nidx, right_nidx) + 1)));

    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(
        nullptr, temp_storage_bytes, position.Current() + segment.begin,
        position.other() + segment.begin, ridx.Current() + segment.begin,
        ridx.other() + segment.begin, segment.Size(), min_bits, max_bits);

    temp_memory.LazyAllocate(temp_storage_bytes);

    hipcub::DeviceRadixSort::SortPairs(
        temp_memory.d_temp_storage, temp_memory.temp_storage_bytes,
        position.Current() + segment.begin, position.other() + segment.begin,
        ridx.Current() + segment.begin, ridx.other() + segment.begin,
        segment.Size(), min_bits, max_bits);
    dh::safe_cuda(hipMemcpy(
        position.Current() + segment.begin, position.other() + segment.begin,
        segment.Size() * sizeof(int), hipMemcpyDeviceToDevice));
    dh::safe_cuda(hipMemcpy(
        ridx.Current() + segment.begin, ridx.other() + segment.begin,
        segment.Size() * sizeof(bst_uint), hipMemcpyDeviceToDevice));
  }

  void UpdatePredictionCache(bst_float* out_preds_d) {
    dh::safe_cuda(hipSetDevice(device_idx));
    if (!prediction_cache_initialised) {
      dh::safe_cuda(hipMemcpy(
          prediction_cache.Data(), out_preds_d,
          prediction_cache.Size() * sizeof(bst_float), hipMemcpyDefault));
    }
    prediction_cache_initialised = true;

    CalcWeightTrainParam param_d(param);

    dh::safe_cuda(hipMemcpy(node_sum_gradients_d.Data(),
                             node_sum_gradients.data(),
                             sizeof(GradientPair) * node_sum_gradients.size(),
                             hipMemcpyHostToDevice));
    auto d_position = position.Current();
    auto d_ridx = ridx.Current();
    auto d_node_sum_gradients = node_sum_gradients_d.Data();
    auto d_prediction_cache = prediction_cache.Data();

    dh::LaunchN(
        device_idx, prediction_cache.Size(), [=] __device__(int local_idx) {
          int pos = d_position[local_idx];
          bst_float weight = CalcWeight(param_d, d_node_sum_gradients[pos]);
          d_prediction_cache[d_ridx[local_idx]] +=
              weight * param_d.learning_rate;
        });

    dh::safe_cuda(hipMemcpy(
        out_preds_d, prediction_cache.Data(),
        prediction_cache.Size() * sizeof(bst_float), hipMemcpyDefault));
  }
};

class GPUHistMaker : public TreeUpdater {
 public:
  struct ExpandEntry;

  GPUHistMaker() : initialised_(false), p_last_fmat_(nullptr) {}
  void Init(
      const std::vector<std::pair<std::string, std::string>>& args) override {
    param_.InitAllowUnknown(args);
    CHECK(param_.n_gpus != 0) << "Must have at least one device";
    n_devices_ = param_.n_gpus;
    devices_ = GPUSet::Range(param_.gpu_id, dh::NDevicesAll(param_.n_gpus));

    dh::CheckComputeCapability();

    if (param_.grow_policy == TrainParam::kLossGuide) {
      qexpand_.reset(new ExpandQueue(LossGuide));
    } else {
      qexpand_.reset(new ExpandQueue(DepthWise));
    }

    monitor_.Init("updater_gpu_hist", param_.debug_verbose);
  }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) override {
    monitor_.Start("Update", device_list_);
    GradStats::CheckInfo(dmat->Info());
    // rescale learning rate according to size of trees
    float lr = param_.learning_rate;
    param_.learning_rate = lr / trees.size();
    ValueConstraint::Init(&param_, dmat->Info().num_col_);
    // build tree
    try {
      for (size_t i = 0; i < trees.size(); ++i) {
        this->UpdateTree(gpair, dmat, trees[i]);
      }
      dh::safe_cuda(hipGetLastError());
    } catch (const std::exception& e) {
      LOG(FATAL) << "Exception in gpu_hist: " << e.what() << std::endl;
    }
    param_.learning_rate = lr;
    monitor_.Stop("Update", device_list_);
  }

  void InitDataOnce(DMatrix* dmat) {
    info_ = &dmat->Info();

    int n_devices = dh::NDevices(param_.n_gpus, info_->num_row_);

    device_list_.resize(n_devices);
    for (int d_idx = 0; d_idx < n_devices; ++d_idx) {
      int device_idx = (param_.gpu_id + d_idx) % dh::NVisibleDevices();
      device_list_[d_idx] = device_idx;
    }

    reducer_.Init(device_list_);

    // Partition input matrix into row segments
    std::vector<size_t> row_segments;
    dh::RowSegments(info_->num_row_, n_devices, &row_segments);

    dmlc::DataIter<SparsePage>* iter = dmat->RowIterator();
    iter->BeforeFirst();
    CHECK(iter->Next()) << "Empty batches are not supported";
    const SparsePage& batch = iter->Value();
    // Create device shards
    shards_.resize(n_devices);
    dh::ExecuteIndexShards(&shards_, [&](int i, std::unique_ptr<DeviceShard>& shard) {
        shard = std::unique_ptr<DeviceShard>
          (new DeviceShard(device_list_[i], i,
                           row_segments[i], row_segments[i + 1], param_));
        shard->InitRowPtrs(batch);
      });

    monitor_.Start("Quantiles", device_list_);
    common::DeviceSketch(batch, *info_, param_, &hmat_);
    n_bins_ = hmat_.row_ptr.back();
    monitor_.Stop("Quantiles", device_list_);

    monitor_.Start("BinningCompression", device_list_);
    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->InitCompressedData(hmat_, batch);
      });
    monitor_.Stop("BinningCompression", device_list_);

    CHECK(!iter->Next()) << "External memory not supported";

    p_last_fmat_ = dmat;
    initialised_ = true;
  }

  void InitData(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
                const RegTree& tree) {
    monitor_.Start("InitDataOnce", device_list_);
    if (!initialised_) {
      this->InitDataOnce(dmat);
    }
    monitor_.Stop("InitDataOnce", device_list_);

    column_sampler_.Init(info_->num_col_, param_);

    // Copy gpair & reset memory
    monitor_.Start("InitDataReset", device_list_);

    gpair->Reshard(devices_);
    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {shard->Reset(gpair); });
    monitor_.Stop("InitDataReset", device_list_);
  }

  void AllReduceHist(int nidx) {
    reducer_.GroupStart();
    for (auto& shard : shards_) {
      auto d_node_hist = shard->hist.GetHistPtr(nidx);
      reducer_.AllReduceSum(
          shard->normalised_device_idx,
          reinterpret_cast<GradientPairSumT::ValueT*>(d_node_hist),
          reinterpret_cast<GradientPairSumT::ValueT*>(d_node_hist),
          n_bins_ * (sizeof(GradientPairSumT) / sizeof(GradientPairSumT::ValueT)));
    }
    reducer_.GroupEnd();

    reducer_.Synchronize();
  }

  void BuildHistLeftRight(int nidx_parent, int nidx_left, int nidx_right) {
    size_t left_node_max_elements = 0;
    size_t right_node_max_elements = 0;
    for (auto& shard : shards_) {
      left_node_max_elements = (std::max)(
          left_node_max_elements, shard->ridx_segments[nidx_left].Size());
      right_node_max_elements = (std::max)(
          right_node_max_elements, shard->ridx_segments[nidx_right].Size());
    }

    auto build_hist_nidx = nidx_left;
    auto subtraction_trick_nidx = nidx_right;

    if (right_node_max_elements < left_node_max_elements) {
      build_hist_nidx = nidx_right;
      subtraction_trick_nidx = nidx_left;
    }

    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->BuildHist(build_hist_nidx);
      });

    this->AllReduceHist(build_hist_nidx);

    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->SubtractionTrick(nidx_parent, build_hist_nidx,
                               subtraction_trick_nidx);
      });
  }

  // Returns best loss
  std::vector<DeviceSplitCandidate> EvaluateSplits(
      const std::vector<int>& nidx_set, RegTree* p_tree) {
    auto columns = info_->num_col_;
    std::vector<DeviceSplitCandidate> best_splits(nidx_set.size());
    std::vector<DeviceSplitCandidate> candidate_splits(nidx_set.size() *
                                                       columns);
    // Use first device
    auto& shard = shards_.front();
    dh::safe_cuda(hipSetDevice(shard->device_idx));
    shard->temp_memory.LazyAllocate(sizeof(DeviceSplitCandidate) * columns *
                                    nidx_set.size());
    auto d_split = shard->temp_memory.Pointer<DeviceSplitCandidate>();

    auto& streams = shard->GetStreams(static_cast<int>(nidx_set.size()));

    // Use streams to process nodes concurrently
    for (auto i = 0; i < nidx_set.size(); i++) {
      auto nidx = nidx_set[i];
      DeviceNodeStats node(shard->node_sum_gradients[nidx], nidx, param_);

      const int BLOCK_THREADS = 256;
      evaluate_split_kernel<BLOCK_THREADS>
          <<<uint32_t(columns), BLOCK_THREADS, 0, streams[i]>>>(
              shard->hist.GetHistPtr(nidx), nidx, info_->num_col_, node,
              shard->feature_segments.Data(), shard->min_fvalue.Data(),
              shard->gidx_fvalue_map.Data(), GPUTrainingParam(param_),
              d_split + i * columns, node_value_constraints_[nidx],
              shard->monotone_constraints.Data());
    }

    dh::safe_cuda(
        hipMemcpy(candidate_splits.data(), shard->temp_memory.d_temp_storage,
                   sizeof(DeviceSplitCandidate) * columns * nidx_set.size(),
                   hipMemcpyDeviceToHost));

    for (auto i = 0; i < nidx_set.size(); i++) {
      auto nidx = nidx_set[i];
      DeviceSplitCandidate nidx_best;
      for (auto fidx = 0; fidx < columns; fidx++) {
        auto& candidate = candidate_splits[i * columns + fidx];
        if (column_sampler_.ColumnUsed(candidate.findex,
                                      p_tree->GetDepth(nidx))) {
          nidx_best.Update(candidate_splits[i * columns + fidx], param_);
        }
      }
      best_splits[i] = nidx_best;
    }
    return std::move(best_splits);
  }

  void InitRoot(RegTree* p_tree) {
    auto root_nidx = 0;
    // Sum gradients
    std::vector<GradientPair> tmp_sums(shards_.size());

    dh::ExecuteIndexShards(&shards_, [&](int i, std::unique_ptr<DeviceShard>& shard) {
        dh::safe_cuda(hipSetDevice(shard->device_idx));
      tmp_sums[i] =
        dh::SumReduction(shard->temp_memory, shard->gpair.Data(),
                         shard->gpair.Size());
      });
    auto sum_gradient =
        std::accumulate(tmp_sums.begin(), tmp_sums.end(), GradientPair());

    // Generate root histogram
    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->BuildHist(root_nidx);
      });

    this->AllReduceHist(root_nidx);

    // Remember root stats
    p_tree->Stat(root_nidx).sum_hess = sum_gradient.GetHess();
    auto weight = CalcWeight(param_, sum_gradient);
    p_tree->Stat(root_nidx).base_weight = weight;
    (*p_tree)[root_nidx].SetLeaf(param_.learning_rate * weight);

    // Store sum gradients
    for (auto& shard : shards_) {
      shard->node_sum_gradients[root_nidx] = sum_gradient;
    }

    // Initialise root constraint
    node_value_constraints_.resize(p_tree->GetNodes().size());

    // Generate first split
    auto splits = this->EvaluateSplits({root_nidx}, p_tree);
    qexpand_->push(
        ExpandEntry(root_nidx, p_tree->GetDepth(root_nidx), splits.front(), 0));
  }

  void UpdatePosition(const ExpandEntry& candidate, RegTree* p_tree) {
    auto nidx = candidate.nid;
    auto left_nidx = (*p_tree)[nidx].LeftChild();
    auto right_nidx = (*p_tree)[nidx].RightChild();

    // convert floating-point split_pt into corresponding bin_id
    // split_cond = -1 indicates that split_pt is less than all known cut points
    auto split_gidx = -1;
    auto fidx = candidate.split.findex;
    auto default_dir_left = candidate.split.dir == kLeftDir;
    auto fidx_begin = hmat_.row_ptr[fidx];
    auto fidx_end = hmat_.row_ptr[fidx + 1];
    for (auto i = fidx_begin; i < fidx_end; ++i) {
      if (candidate.split.fvalue == hmat_.cut[i]) {
        split_gidx = static_cast<int32_t>(i);
      }
    }

    auto is_dense = info_->num_nonzero_ == info_->num_row_ * info_->num_col_;

    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
      shard->UpdatePosition(nidx, left_nidx, right_nidx, fidx,
                           split_gidx, default_dir_left,
                           is_dense, fidx_begin, fidx_end);
      });
  }

  void ApplySplit(const ExpandEntry& candidate, RegTree* p_tree) {
    // Add new leaves
    RegTree& tree = *p_tree;
    tree.AddChilds(candidate.nid);
    auto& parent = tree[candidate.nid];
    parent.SetSplit(candidate.split.findex, candidate.split.fvalue,
                     candidate.split.dir == kLeftDir);
    tree.Stat(candidate.nid).loss_chg = candidate.split.loss_chg;

    // Set up child constraints
    node_value_constraints_.resize(tree.GetNodes().size());
    GradStats left_stats(param_);
    left_stats.Add(candidate.split.left_sum);
    GradStats right_stats(param_);
    right_stats.Add(candidate.split.right_sum);
    node_value_constraints_[candidate.nid].SetChild(
        param_, parent.SplitIndex(), left_stats, right_stats,
        &node_value_constraints_[parent.LeftChild()],
        &node_value_constraints_[parent.RightChild()]);

    // Configure left child
    auto left_weight =
        node_value_constraints_[parent.LeftChild()].CalcWeight(param_, left_stats);
    tree[parent.LeftChild()].SetLeaf(left_weight * param_.learning_rate, 0);
    tree.Stat(parent.LeftChild()).base_weight = left_weight;
    tree.Stat(parent.LeftChild()).sum_hess = candidate.split.left_sum.GetHess();

    // Configure right child
    auto right_weight =
        node_value_constraints_[parent.RightChild()].CalcWeight(param_, right_stats);
    tree[parent.RightChild()].SetLeaf(right_weight * param_.learning_rate, 0);
    tree.Stat(parent.RightChild()).base_weight = right_weight;
    tree.Stat(parent.RightChild()).sum_hess = candidate.split.right_sum.GetHess();
    // Store sum gradients
    for (auto& shard : shards_) {
      shard->node_sum_gradients[parent.LeftChild()] = candidate.split.left_sum;
      shard->node_sum_gradients[parent.RightChild()] = candidate.split.right_sum;
    }
    this->UpdatePosition(candidate, p_tree);
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* p_fmat,
                  RegTree* p_tree) {
    auto& tree = *p_tree;

    monitor_.Start("InitData", device_list_);
    this->InitData(gpair, p_fmat, *p_tree);
    monitor_.Stop("InitData", device_list_);
    monitor_.Start("InitRoot", device_list_);
    this->InitRoot(p_tree);
    monitor_.Stop("InitRoot", device_list_);

    auto timestamp = qexpand_->size();
    auto num_leaves = 1;

    while (!qexpand_->empty()) {
      auto candidate = qexpand_->top();
      qexpand_->pop();
      if (!candidate.IsValid(param_, num_leaves)) continue;
      // std::cout << candidate;
      monitor_.Start("ApplySplit", device_list_);
      this->ApplySplit(candidate, p_tree);
      monitor_.Stop("ApplySplit", device_list_);
      num_leaves++;

      auto left_child_nidx = tree[candidate.nid].LeftChild();
      auto right_child_nidx = tree[candidate.nid].RightChild();

      // Only create child entries if needed
      if (ExpandEntry::ChildIsValid(param_, tree.GetDepth(left_child_nidx),
                                    num_leaves)) {
        monitor_.Start("BuildHist", device_list_);
        this->BuildHistLeftRight(candidate.nid, left_child_nidx,
                                 right_child_nidx);
        monitor_.Stop("BuildHist", device_list_);

        monitor_.Start("EvaluateSplits", device_list_);
        auto splits =
            this->EvaluateSplits({left_child_nidx, right_child_nidx}, p_tree);
        qexpand_->push(ExpandEntry(left_child_nidx,
                                   tree.GetDepth(left_child_nidx), splits[0],
                                   timestamp++));
        qexpand_->push(ExpandEntry(right_child_nidx,
                                   tree.GetDepth(right_child_nidx), splits[1],
                                   timestamp++));
        monitor_.Stop("EvaluateSplits", device_list_);
      }
    }
  }

  bool UpdatePredictionCache(
      const DMatrix* data, HostDeviceVector<bst_float>* p_out_preds) override {
    monitor_.Start("UpdatePredictionCache", device_list_);
    if (shards_.empty() || p_last_fmat_ == nullptr || p_last_fmat_ != data)
      return false;
    p_out_preds->Reshard(devices_);
    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->UpdatePredictionCache(p_out_preds->DevicePointer(shard->device_idx));
      });
    monitor_.Stop("UpdatePredictionCache", device_list_);
    return true;
  }

  struct ExpandEntry {
    int nid;
    int depth;
    DeviceSplitCandidate split;
    uint64_t timestamp;
    ExpandEntry(int nid, int depth, const DeviceSplitCandidate& split,
                uint64_t timestamp)
        : nid(nid), depth(depth), split(split), timestamp(timestamp) {}
    bool IsValid(const TrainParam& param, int num_leaves) const {
      if (split.loss_chg <= kRtEps) return false;
      if (split.left_sum.GetHess() == 0 || split.right_sum.GetHess() == 0)
        return false;
      if (param.max_depth > 0 && depth == param.max_depth) return false;
      if (param.max_leaves > 0 && num_leaves == param.max_leaves) return false;
      return true;
    }

    static bool ChildIsValid(const TrainParam& param, int depth,
                             int num_leaves) {
      if (param.max_depth > 0 && depth == param.max_depth) return false;
      if (param.max_leaves > 0 && num_leaves == param.max_leaves) return false;
      return true;
    }

    friend std::ostream& operator<<(std::ostream& os, const ExpandEntry& e) {
      os << "ExpandEntry: \n";
      os << "nidx: " << e.nid << "\n";
      os << "depth: " << e.depth << "\n";
      os << "loss: " << e.split.loss_chg << "\n";
      os << "left_sum: " << e.split.left_sum << "\n";
      os << "right_sum: " << e.split.right_sum << "\n";
      return os;
    }
  };

  inline static bool DepthWise(ExpandEntry lhs, ExpandEntry rhs) {
    if (lhs.depth == rhs.depth) {
      return lhs.timestamp > rhs.timestamp;  // favor small timestamp
    } else {
      return lhs.depth > rhs.depth;  // favor small depth
    }
  }
  inline static bool LossGuide(ExpandEntry lhs, ExpandEntry rhs) {
    if (lhs.split.loss_chg == rhs.split.loss_chg) {
      return lhs.timestamp > rhs.timestamp;  // favor small timestamp
    } else {
      return lhs.split.loss_chg < rhs.split.loss_chg;  // favor large loss_chg
    }
  }
  TrainParam param_;
  common::HistCutMatrix hmat_;
  common::GHistIndexMatrix gmat_;
  MetaInfo* info_;
  bool initialised_;
  int n_devices_;
  int n_bins_;

  std::vector<std::unique_ptr<DeviceShard>> shards_;
  ColumnSampler column_sampler_;
  typedef std::priority_queue<ExpandEntry, std::vector<ExpandEntry>,
                              std::function<bool(ExpandEntry, ExpandEntry)>>
      ExpandQueue;
  std::unique_ptr<ExpandQueue> qexpand_;
  common::Monitor monitor_;
  dh::AllReducer reducer_;
  std::vector<ValueConstraint> node_value_constraints_;
  std::vector<int> device_list_;

  DMatrix* p_last_fmat_;
  GPUSet devices_;
};

XGBOOST_REGISTER_TREE_UPDATER(GPUHistMaker, "grow_gpu_hist")
    .describe("Grow tree with GPU.")
    .set_body([]() { return new GPUHistMaker(); });
}  // namespace tree
}  // namespace xgboost
