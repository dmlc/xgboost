#include "hip/hip_runtime.h"
/*!
 * Copyright 2017 XGBoost contributors
 */
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <xgboost/tree_updater.h>
#include <algorithm>
#include <memory>
#include <queue>
#include <utility>
#include <vector>
#include "../common/compressed_iterator.h"
#include "../common/device_helpers.cuh"
#include "../common/hist_util.h"
#include "../common/host_device_vector.h"
#include "../common/timer.h"
#include "param.h"
#include "updater_gpu_common.cuh"

namespace xgboost {
namespace tree {

DMLC_REGISTRY_FILE_TAG(updater_gpu_hist);

using GradientPairSumT = GradientPairPrecise;

template <int BLOCK_THREADS, typename ReduceT, typename TempStorageT>
__device__ GradientPairSumT ReduceFeature(const GradientPairSumT* begin,
                                     const GradientPairSumT* end,
                                     TempStorageT* temp_storage) {
  __shared__ hipcub::Uninitialized<GradientPairSumT> uninitialized_sum;
  GradientPairSumT& shared_sum = uninitialized_sum.Alias();

  GradientPairSumT local_sum = GradientPairSumT();
  for (auto itr = begin; itr < end; itr += BLOCK_THREADS) {
    bool thread_active = itr + threadIdx.x < end;
    // Scan histogram
    GradientPairSumT bin = thread_active ? *(itr + threadIdx.x) : GradientPairSumT();

    local_sum += ReduceT(temp_storage->sum_reduce).Reduce(bin, hipcub::Sum());
  }

  if (threadIdx.x == 0) {
    shared_sum = local_sum;
  }
  __syncthreads();

  return shared_sum;
}

template <int BLOCK_THREADS, typename ReduceT, typename scan_t,
          typename max_ReduceT, typename TempStorageT>
__device__ void EvaluateFeature(int fidx, const GradientPairSumT* hist,
                                const int* feature_segments, float min_fvalue,
                                const float* gidx_fvalue_map,
                                DeviceSplitCandidate* best_split,
                                const DeviceNodeStats& node,
                                const GPUTrainingParam& param,
                                TempStorageT* temp_storage, int constraint,
                                const ValueConstraint& value_constraint) {
  int gidx_begin = feature_segments[fidx];
  int gidx_end = feature_segments[fidx + 1];

  GradientPairSumT feature_sum = ReduceFeature<BLOCK_THREADS, ReduceT>(
      hist + gidx_begin, hist + gidx_end, temp_storage);

  auto prefix_op = SumCallbackOp<GradientPairSumT>();
  for (int scan_begin = gidx_begin; scan_begin < gidx_end;
       scan_begin += BLOCK_THREADS) {
    bool thread_active = scan_begin + threadIdx.x < gidx_end;

    GradientPairSumT bin =
        thread_active ? hist[scan_begin + threadIdx.x] : GradientPairSumT();
    scan_t(temp_storage->scan).ExclusiveScan(bin, bin, hipcub::Sum(), prefix_op);

    // Calculate  gain
    GradientPairSumT parent_sum = GradientPairSumT(node.sum_gradients);

    GradientPairSumT missing = parent_sum - feature_sum;

    bool missing_left = true;
    const float null_gain = -FLT_MAX;
    float gain = null_gain;
    if (thread_active) {
      gain = LossChangeMissing(bin, missing, parent_sum, node.root_gain, param,
                              constraint, value_constraint, missing_left);
    }

    __syncthreads();

    // Find thread with best gain
    hipcub::KeyValuePair<int, float> tuple(threadIdx.x, gain);
    hipcub::KeyValuePair<int, float> best =
        max_ReduceT(temp_storage->max_reduce).Reduce(tuple, hipcub::ArgMax());

    __shared__ hipcub::KeyValuePair<int, float> block_max;
    if (threadIdx.x == 0) {
      block_max = best;
    }

    __syncthreads();

    // Best thread updates split
    if (threadIdx.x == block_max.key) {
      int gidx = scan_begin + threadIdx.x;
      float fvalue =
          gidx == gidx_begin ? min_fvalue : gidx_fvalue_map[gidx - 1];

      GradientPairSumT left = missing_left ? bin + missing : bin;
      GradientPairSumT right = parent_sum - left;

      best_split->Update(gain, missing_left ? kLeftDir : kRightDir, fvalue, fidx,
                         GradientPair(left), GradientPair(right), param);
    }
    __syncthreads();
  }
}

template <int BLOCK_THREADS>
__global__ void evaluate_split_kernel(
    const GradientPairSumT* d_hist, int nidx, uint64_t n_features,
    DeviceNodeStats nodes, const int* d_feature_segments,
    const float* d_fidx_min_map, const float* d_gidx_fvalue_map,
    GPUTrainingParam gpu_param, DeviceSplitCandidate* d_split,
    ValueConstraint value_constraint, int* d_monotonic_constraints) {
  typedef hipcub::KeyValuePair<int, float> ArgMaxT;
  typedef hipcub::BlockScan<GradientPairSumT, BLOCK_THREADS, hipcub::BLOCK_SCAN_WARP_SCANS>
      BlockScanT;
  typedef hipcub::BlockReduce<ArgMaxT, BLOCK_THREADS> MaxReduceT;

  typedef hipcub::BlockReduce<GradientPairSumT, BLOCK_THREADS> SumReduceT;

  union TempStorage {
    typename BlockScanT::TempStorage scan;
    typename MaxReduceT::TempStorage max_reduce;
    typename SumReduceT::TempStorage sum_reduce;
  };

  __shared__ hipcub::Uninitialized<DeviceSplitCandidate> uninitialized_split;
  DeviceSplitCandidate& best_split = uninitialized_split.Alias();
  __shared__ TempStorage temp_storage;

  if (threadIdx.x == 0) {
    best_split = DeviceSplitCandidate();
  }

  __syncthreads();

  auto fidx = blockIdx.x;
  auto constraint = d_monotonic_constraints[fidx];
  EvaluateFeature<BLOCK_THREADS, SumReduceT, BlockScanT, MaxReduceT>(
      fidx, d_hist, d_feature_segments, d_fidx_min_map[fidx], d_gidx_fvalue_map,
      &best_split, nodes, gpu_param, &temp_storage, constraint,
      value_constraint);

  __syncthreads();

  if (threadIdx.x == 0) {
    // Record best loss
    d_split[fidx] = best_split;
  }
}

// Find a gidx value for a given feature otherwise return -1 if not found
template <typename GidxIterT>
__device__ int BinarySearchRow(bst_uint begin, bst_uint end, GidxIterT data,
                               int fidx_begin, int fidx_end) {
  bst_uint previous_middle = UINT32_MAX;
  while (end != begin) {
    auto middle = begin + (end - begin) / 2;
    if (middle == previous_middle) {
      break;
    }
    previous_middle = middle;

    auto gidx = data[middle];

    if (gidx >= fidx_begin && gidx < fidx_end) {
      return gidx;
    } else if (gidx < fidx_begin) {
      begin = middle;
    } else {
      end = middle;
    }
  }
  // Value is missing
  return -1;
}

struct DeviceHistogram {
  dh::BulkAllocator<dh::MemoryType::kDevice> ba;
  dh::DVec<GradientPairSumT> data;
  int n_bins;
  void Init(int device_idx, int max_nodes, int n_bins, bool silent) {
    this->n_bins = n_bins;
    ba.Allocate(device_idx, silent, &data, size_t(max_nodes) * size_t(n_bins));
  }

  void Reset() { data.Fill(GradientPairSumT()); }
  GradientPairSumT* GetHistPtr(int nidx) { return data.Data() + nidx * n_bins; }

  void PrintNidx(int nidx) const {
    auto h_data = data.AsVector();
    std::cout << "nidx " << nidx << ":\n";
    for (int i = n_bins * nidx; i < n_bins * (nidx + 1); i++) {
      std::cout << h_data[i] << " ";
    }
    std::cout << "\n";
  }
};

struct CalcWeightTrainParam {
  float min_child_weight;
  float reg_alpha;
  float reg_lambda;
  float max_delta_step;
  float learning_rate;
  XGBOOST_DEVICE explicit CalcWeightTrainParam(const TrainParam& p)
      : min_child_weight(p.min_child_weight),
        reg_alpha(p.reg_alpha),
        reg_lambda(p.reg_lambda),
        max_delta_step(p.max_delta_step),
        learning_rate(p.learning_rate) {}
};

// Manage memory for a single GPU
struct DeviceShard {
  struct Segment {
    size_t begin;
    size_t end;

    Segment() : begin(0), end(0) {}

    Segment(size_t begin, size_t end) : begin(begin), end(end) {
      CHECK_GE(end, begin);
    }
    size_t Size() const { return end - begin; }
  };

  int device_idx;
  int normalised_device_idx;  // Device index counting from param.gpu_id
  dh::BulkAllocator<dh::MemoryType::kDevice> ba;
  dh::DVec<common::CompressedByteT> gidx_buffer;
  dh::DVec<GradientPair> gpair;
  dh::DVec2<bst_uint> ridx;  // Row index relative to this shard
  dh::DVec2<int> position;
  std::vector<Segment> ridx_segments;
  dh::DVec<int> feature_segments;
  dh::DVec<float> gidx_fvalue_map;
  dh::DVec<float> min_fvalue;
  dh::DVec<int> monotone_constraints;
  dh::DVec<bst_float> prediction_cache;
  std::vector<GradientPair> node_sum_gradients;
  dh::DVec<GradientPair> node_sum_gradients_d;
  common::CompressedIterator<uint32_t> gidx;
  int row_stride;
  bst_uint row_begin_idx;  // The row offset for this shard
  bst_uint row_end_idx;
  bst_uint n_rows;
  int n_bins;
  int null_gidx_value;
  DeviceHistogram hist;
  TrainParam param;
  bool prediction_cache_initialised;

  int64_t* tmp_pinned;  // Small amount of staging memory

  std::vector<hipStream_t> streams;

  dh::CubMemory temp_memory;

  DeviceShard(int device_idx, int normalised_device_idx,
              const common::GHistIndexMatrix& gmat, bst_uint row_begin,
              bst_uint row_end, int n_bins, TrainParam param)
      : device_idx(device_idx),
        normalised_device_idx(normalised_device_idx),
        row_begin_idx(row_begin),
        row_end_idx(row_end),
        n_rows(row_end - row_begin),
        n_bins(n_bins),
        null_gidx_value(n_bins),
        param(param),
        prediction_cache_initialised(false) {
    // Convert to ELLPACK matrix representation
    int max_elements_row = 0;
    for (auto i = row_begin; i < row_end; i++) {
      max_elements_row =
          (std::max)(max_elements_row,
                     static_cast<int>(gmat.row_ptr[i + 1] - gmat.row_ptr[i]));
    }
    row_stride = max_elements_row;
    std::vector<int> ellpack_matrix(row_stride * n_rows, null_gidx_value);

    for (auto i = row_begin; i < row_end; i++) {
      int row_count = 0;
      for (auto j = gmat.row_ptr[i]; j < gmat.row_ptr[i + 1]; j++) {
        ellpack_matrix[(i - row_begin) * row_stride + row_count] =
            gmat.index[j];
        row_count++;
      }
    }

    // Allocate
    int num_symbols = n_bins + 1;
    size_t compressed_size_bytes =
        common::CompressedBufferWriter::CalculateBufferSize(
            ellpack_matrix.size(), num_symbols);

    CHECK(!(param.max_leaves == 0 && param.max_depth == 0))
        << "Max leaves and max depth cannot both be unconstrained for "
           "gpu_hist.";
    int max_nodes =
        param.max_leaves > 0 ? param.max_leaves * 2 : MaxNodesDepth(param.max_depth);
    ba.Allocate(device_idx, param.silent, &gidx_buffer, compressed_size_bytes,
                &gpair, n_rows, &ridx, n_rows, &position, n_rows,
                &prediction_cache, n_rows, &node_sum_gradients_d, max_nodes,
                &feature_segments, gmat.cut->row_ptr.size(), &gidx_fvalue_map,
                gmat.cut->cut.size(), &min_fvalue, gmat.cut->min_val.size(),
                &monotone_constraints, param.monotone_constraints.size());
    gidx_fvalue_map = gmat.cut->cut;
    min_fvalue = gmat.cut->min_val;
    feature_segments = gmat.cut->row_ptr;
    monotone_constraints = param.monotone_constraints;

    node_sum_gradients.resize(max_nodes);
    ridx_segments.resize(max_nodes);

    // Compress gidx
    common::CompressedBufferWriter cbw(num_symbols);
    std::vector<common::CompressedByteT> host_buffer(gidx_buffer.Size());
    cbw.Write(host_buffer.data(), ellpack_matrix.begin(), ellpack_matrix.end());
    gidx_buffer = host_buffer;
    gidx =
        common::CompressedIterator<uint32_t>(gidx_buffer.Data(), num_symbols);

    common::CompressedIterator<uint32_t> ci_host(host_buffer.data(),
                                                 num_symbols);

    // Init histogram
    hist.Init(device_idx, max_nodes, gmat.cut->row_ptr.back(), param.silent);

    dh::safe_cuda(hipHostMalloc(&tmp_pinned, sizeof(int64_t)));
  }

  ~DeviceShard() {
    for (auto& stream : streams) {
      dh::safe_cuda(hipStreamDestroy(stream));
    }
    dh::safe_cuda(hipHostFree(tmp_pinned));
  }

  // Get vector of at least n initialised streams
  std::vector<hipStream_t>& GetStreams(int n) {
    if (n > streams.size()) {
      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamDestroy(stream));
      }

      streams.clear();
      streams.resize(n);

      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamCreate(&stream));
      }
    }

    return streams;
  }

  // Reset values for each update iteration
  void Reset(HostDeviceVector<GradientPair>* dh_gpair, int device) {
    auto begin = dh_gpair->tbegin(device);
    dh::safe_cuda(hipSetDevice(device_idx));
    position.CurrentDVec().Fill(0);
    std::fill(node_sum_gradients.begin(), node_sum_gradients.end(),
              GradientPair());

    thrust::sequence(ridx.CurrentDVec().tbegin(), ridx.CurrentDVec().tend());

    std::fill(ridx_segments.begin(), ridx_segments.end(), Segment(0, 0));
    ridx_segments.front() = Segment(0, ridx.Size());
    this->gpair.copy(begin + row_begin_idx, begin + row_end_idx);
    SubsampleGradientPair(&gpair, param.subsample, row_begin_idx);
    hist.Reset();
  }

  void BuildHist(int nidx) {
    auto segment = ridx_segments[nidx];
    auto d_node_hist = hist.GetHistPtr(nidx);
    auto d_gidx = gidx;
    auto d_ridx = ridx.Current();
    auto d_gpair = gpair.Data();
    auto row_stride = this->row_stride;
    auto null_gidx_value = this->null_gidx_value;
    auto n_elements = segment.Size() * row_stride;

    dh::LaunchN(device_idx, n_elements, [=] __device__(size_t idx) {
      int ridx = d_ridx[(idx / row_stride) + segment.begin];
      int gidx = d_gidx[ridx * row_stride + idx % row_stride];

      if (gidx != null_gidx_value) {
        AtomicAddGpair(d_node_hist + gidx, d_gpair[ridx]);
      }
    });
  }
  void SubtractionTrick(int nidx_parent, int nidx_histogram,
                        int nidx_subtraction) {
    auto d_node_hist_parent = hist.GetHistPtr(nidx_parent);
    auto d_node_hist_histogram = hist.GetHistPtr(nidx_histogram);
    auto d_node_hist_subtraction = hist.GetHistPtr(nidx_subtraction);

    dh::LaunchN(device_idx, hist.n_bins, [=] __device__(size_t idx) {
      d_node_hist_subtraction[idx] =
          d_node_hist_parent[idx] - d_node_hist_histogram[idx];
    });
  }

  __device__ void CountLeft(int64_t* d_count, int val, int left_nidx) {
    unsigned ballot = __ballot(val == left_nidx);
    if (threadIdx.x % 32 == 0) {
      atomicAdd(reinterpret_cast<unsigned long long*>(d_count),    // NOLINT
                static_cast<unsigned long long>(__popc(ballot)));  // NOLINT
    }
  }

  void UpdatePosition(int nidx, int left_nidx, int right_nidx, int fidx,
                      int split_gidx, bool default_dir_left, bool is_dense,
                      int fidx_begin, int fidx_end) {
    dh::safe_cuda(hipSetDevice(device_idx));
    temp_memory.LazyAllocate(sizeof(int64_t));
    auto d_left_count = temp_memory.Pointer<int64_t>();
    dh::safe_cuda(hipMemset(d_left_count, 0, sizeof(int64_t)));
    auto segment = ridx_segments[nidx];
    auto d_ridx = ridx.Current();
    auto d_position = position.Current();
    auto d_gidx = gidx;
    auto row_stride = this->row_stride;
    dh::LaunchN<1, 512>(
        device_idx, segment.Size(), [=] __device__(bst_uint idx) {
          idx += segment.begin;
          auto ridx = d_ridx[idx];
          auto row_begin = row_stride * ridx;
          auto row_end = row_begin + row_stride;
          auto gidx = -1;
          if (is_dense) {
            gidx = d_gidx[row_begin + fidx];
          } else {
            gidx = BinarySearchRow(row_begin, row_end, d_gidx, fidx_begin,
                                   fidx_end);
          }

          int position;
          if (gidx >= 0) {
            // Feature is found
            position = gidx <= split_gidx ? left_nidx : right_nidx;
          } else {
            // Feature is missing
            position = default_dir_left ? left_nidx : right_nidx;
          }

          CountLeft(d_left_count, position, left_nidx);
          d_position[idx] = position;
        });

    dh::safe_cuda(hipMemcpy(tmp_pinned, d_left_count, sizeof(int64_t),
                             hipMemcpyDeviceToHost));
    auto left_count = *tmp_pinned;

    SortPosition(segment, left_nidx, right_nidx);
    // dh::safe_cuda(hipStreamSynchronize(stream));
    ridx_segments[left_nidx] =
        Segment(segment.begin, segment.begin + left_count);
    ridx_segments[right_nidx] =
        Segment(segment.begin + left_count, segment.end);
  }

  void SortPosition(const Segment& segment, int left_nidx, int right_nidx) {
    int min_bits = 0;
    int max_bits = static_cast<int>(
        std::ceil(std::log2((std::max)(left_nidx, right_nidx) + 1)));

    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(
        nullptr, temp_storage_bytes, position.Current() + segment.begin,
        position.other() + segment.begin, ridx.Current() + segment.begin,
        ridx.other() + segment.begin, segment.Size(), min_bits, max_bits);

    temp_memory.LazyAllocate(temp_storage_bytes);

    hipcub::DeviceRadixSort::SortPairs(
        temp_memory.d_temp_storage, temp_memory.temp_storage_bytes,
        position.Current() + segment.begin, position.other() + segment.begin,
        ridx.Current() + segment.begin, ridx.other() + segment.begin,
        segment.Size(), min_bits, max_bits);
    dh::safe_cuda(hipMemcpy(
        position.Current() + segment.begin, position.other() + segment.begin,
        segment.Size() * sizeof(int), hipMemcpyDeviceToDevice));
    dh::safe_cuda(hipMemcpy(
        ridx.Current() + segment.begin, ridx.other() + segment.begin,
        segment.Size() * sizeof(bst_uint), hipMemcpyDeviceToDevice));
  }

  void UpdatePredictionCache(bst_float* out_preds_d) {
    dh::safe_cuda(hipSetDevice(device_idx));
    if (!prediction_cache_initialised) {
      dh::safe_cuda(hipMemcpy(
          prediction_cache.Data(), &out_preds_d[row_begin_idx],
          prediction_cache.Size() * sizeof(bst_float), hipMemcpyDefault));
    }
    prediction_cache_initialised = true;

    CalcWeightTrainParam param_d(param);

    thrust::copy(node_sum_gradients.begin(), node_sum_gradients.end(),
                 node_sum_gradients_d.tbegin());
    auto d_position = position.Current();
    auto d_ridx = ridx.Current();
    auto d_node_sum_gradients = node_sum_gradients_d.Data();
    auto d_prediction_cache = prediction_cache.Data();

    dh::LaunchN(
        device_idx, prediction_cache.Size(), [=] __device__(int local_idx) {
          int pos = d_position[local_idx];
          bst_float weight = CalcWeight(param_d, d_node_sum_gradients[pos]);
          d_prediction_cache[d_ridx[local_idx]] +=
              weight * param_d.learning_rate;
        });

    dh::safe_cuda(hipMemcpy(
        &out_preds_d[row_begin_idx], prediction_cache.Data(),
        prediction_cache.Size() * sizeof(bst_float), hipMemcpyDefault));
  }
};

class GPUHistMaker : public TreeUpdater {
 public:
  struct ExpandEntry;

  GPUHistMaker() : initialised_(false), p_last_fmat_(nullptr) {}
  void Init(
      const std::vector<std::pair<std::string, std::string>>& args) override {
    param_.InitAllowUnknown(args);
    CHECK(param_.n_gpus != 0) << "Must have at least one device";
    n_devices_ = param_.n_gpus;

    dh::CheckComputeCapability();

    if (param_.grow_policy == TrainParam::kLossGuide) {
      qexpand_.reset(new ExpandQueue(LossGuide));
    } else {
      qexpand_.reset(new ExpandQueue(DepthWise));
    }

    monitor_.Init("updater_gpu_hist", param_.debug_verbose);
  }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) override {
    monitor_.Start("Update", device_list_);
    GradStats::CheckInfo(dmat->Info());
    // rescale learning rate according to size of trees
    float lr = param_.learning_rate;
    param_.learning_rate = lr / trees.size();
    ValueConstraint::Init(&param_, dmat->Info().num_col_);
    // build tree
    try {
      for (size_t i = 0; i < trees.size(); ++i) {
        this->UpdateTree(gpair, dmat, trees[i]);
      }
    } catch (const std::exception& e) {
      LOG(FATAL) << "GPU plugin exception: " << e.what() << std::endl;
    }
    param_.learning_rate = lr;
    monitor_.Stop("Update", device_list_);
  }

  void InitDataOnce(DMatrix* dmat) {
    info_ = &dmat->Info();
    monitor_.Start("Quantiles", device_list_);
    hmat_.Init(dmat, param_.max_bin);
    gmat_.cut = &hmat_;
    gmat_.Init(dmat);
    monitor_.Stop("Quantiles", device_list_);
    n_bins_ = hmat_.row_ptr.back();

    int n_devices = dh::NDevices(param_.n_gpus, info_->num_row_);

    bst_uint row_begin = 0;
    bst_uint shard_size =
        std::ceil(static_cast<double>(info_->num_row_) / n_devices);

    device_list_.resize(n_devices);
    for (int d_idx = 0; d_idx < n_devices; ++d_idx) {
      int device_idx = (param_.gpu_id + d_idx) % dh::NVisibleDevices();
      device_list_[d_idx] = device_idx;
    }

    reducer_.Init(device_list_);

    // Partition input matrix into row segments
    std::vector<size_t> row_segments;
    shards_.resize(n_devices);
    row_segments.push_back(0);
    for (int d_idx = 0; d_idx < n_devices; ++d_idx) {
      bst_uint row_end =
          std::min(static_cast<size_t>(row_begin + shard_size), info_->num_row_);
      row_segments.push_back(row_end);
      row_begin = row_end;
    }

    // Create device shards
    omp_set_num_threads(shards_.size());
#pragma omp parallel
    {
      auto cpu_thread_id = omp_get_thread_num();
      shards_[cpu_thread_id] = std::unique_ptr<DeviceShard>(
          new DeviceShard(device_list_[cpu_thread_id], cpu_thread_id, gmat_,
                          row_segments[cpu_thread_id],
                          row_segments[cpu_thread_id + 1], n_bins_, param_));
    }

    p_last_fmat_ = dmat;
    initialised_ = true;
  }

  void InitData(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
                const RegTree& tree) {
    monitor_.Start("InitDataOnce", device_list_);
    if (!initialised_) {
      this->InitDataOnce(dmat);
    }
    monitor_.Stop("InitDataOnce", device_list_);

    column_sampler_.Init(info_->num_col_, param_);

    // Copy gpair & reset memory
    monitor_.Start("InitDataReset", device_list_);
    omp_set_num_threads(shards_.size());

    // TODO(canonizer): make it parallel again once HostDeviceVector is
    // thread-safe
    for (int shard = 0; shard < shards_.size(); ++shard)
      shards_[shard]->Reset(gpair, param_.gpu_id);
    monitor_.Stop("InitDataReset", device_list_);
  }

  void AllReduceHist(int nidx) {
    for (auto& shard : shards_) {
      auto d_node_hist = shard->hist.GetHistPtr(nidx);
      reducer_.AllReduceSum(
          shard->normalised_device_idx,
          reinterpret_cast<GradientPairSumT::ValueT*>(d_node_hist),
          reinterpret_cast<GradientPairSumT::ValueT*>(d_node_hist),
          n_bins_ * (sizeof(GradientPairSumT) / sizeof(GradientPairSumT::ValueT)));
    }

    reducer_.Synchronize();
  }

  void BuildHistLeftRight(int nidx_parent, int nidx_left, int nidx_right) {
    size_t left_node_max_elements = 0;
    size_t right_node_max_elements = 0;
    for (auto& shard : shards_) {
      left_node_max_elements = (std::max)(
          left_node_max_elements, shard->ridx_segments[nidx_left].Size());
      right_node_max_elements = (std::max)(
          right_node_max_elements, shard->ridx_segments[nidx_right].Size());
    }

    auto build_hist_nidx = nidx_left;
    auto subtraction_trick_nidx = nidx_right;

    if (right_node_max_elements < left_node_max_elements) {
      build_hist_nidx = nidx_right;
      subtraction_trick_nidx = nidx_left;
    }

    for (auto& shard : shards_) {
      shard->BuildHist(build_hist_nidx);
    }

    this->AllReduceHist(build_hist_nidx);

    for (auto& shard : shards_) {
      shard->SubtractionTrick(nidx_parent, build_hist_nidx,
                              subtraction_trick_nidx);
    }
  }

  // Returns best loss
  std::vector<DeviceSplitCandidate> EvaluateSplits(
      const std::vector<int>& nidx_set, RegTree* p_tree) {
    auto columns = info_->num_col_;
    std::vector<DeviceSplitCandidate> best_splits(nidx_set.size());
    std::vector<DeviceSplitCandidate> candidate_splits(nidx_set.size() *
                                                       columns);
    // Use first device
    auto& shard = shards_.front();
    dh::safe_cuda(hipSetDevice(shard->device_idx));
    shard->temp_memory.LazyAllocate(sizeof(DeviceSplitCandidate) * columns *
                                    nidx_set.size());
    auto d_split = shard->temp_memory.Pointer<DeviceSplitCandidate>();

    auto& streams = shard->GetStreams(static_cast<int>(nidx_set.size()));

    // Use streams to process nodes concurrently
    for (auto i = 0; i < nidx_set.size(); i++) {
      auto nidx = nidx_set[i];
      DeviceNodeStats node(shard->node_sum_gradients[nidx], nidx, param_);

      const int BLOCK_THREADS = 256;
      evaluate_split_kernel<BLOCK_THREADS>
          <<<uint32_t(columns), BLOCK_THREADS, 0, streams[i]>>>(
              shard->hist.GetHistPtr(nidx), nidx, info_->num_col_, node,
              shard->feature_segments.Data(), shard->min_fvalue.Data(),
              shard->gidx_fvalue_map.Data(), GPUTrainingParam(param_),
              d_split + i * columns, node_value_constraints_[nidx],
              shard->monotone_constraints.Data());
    }

    dh::safe_cuda(
        hipMemcpy(candidate_splits.data(), shard->temp_memory.d_temp_storage,
                   sizeof(DeviceSplitCandidate) * columns * nidx_set.size(),
                   hipMemcpyDeviceToHost));

    for (auto i = 0; i < nidx_set.size(); i++) {
      auto nidx = nidx_set[i];
      DeviceSplitCandidate nidx_best;
      for (auto fidx = 0; fidx < columns; fidx++) {
        auto& candidate = candidate_splits[i * columns + fidx];
        if (column_sampler_.ColumnUsed(candidate.findex,
                                      p_tree->GetDepth(nidx))) {
          nidx_best.Update(candidate_splits[i * columns + fidx], param_);
        }
      }
      best_splits[i] = nidx_best;
    }
    return std::move(best_splits);
  }

  void InitRoot(RegTree* p_tree) {
    auto root_nidx = 0;
    // Sum gradients
    std::vector<GradientPair> tmp_sums(shards_.size());
    omp_set_num_threads(shards_.size());
#pragma omp parallel
    {
      auto cpu_thread_id = omp_get_thread_num();
      auto& shard = shards_[cpu_thread_id];
      dh::safe_cuda(hipSetDevice(shard->device_idx));
      tmp_sums[cpu_thread_id] = dh::SumReduction(
          shard->temp_memory, shard->gpair.Data(), shard->gpair.Size());
    }
    auto sum_gradient =
        std::accumulate(tmp_sums.begin(), tmp_sums.end(), GradientPair());

    // Generate root histogram
    for (auto& shard : shards_) {
      shard->BuildHist(root_nidx);
    }

    this->AllReduceHist(root_nidx);

    // Remember root stats
    p_tree->Stat(root_nidx).sum_hess = sum_gradient.GetHess();
    auto weight = CalcWeight(param_, sum_gradient);
    p_tree->Stat(root_nidx).base_weight = weight;
    (*p_tree)[root_nidx].SetLeaf(param_.learning_rate * weight);

    // Store sum gradients
    for (auto& shard : shards_) {
      shard->node_sum_gradients[root_nidx] = sum_gradient;
    }

    // Initialise root constraint
    node_value_constraints_.resize(p_tree->GetNodes().size());

    // Generate first split
    auto splits = this->EvaluateSplits({root_nidx}, p_tree);
    qexpand_->push(
        ExpandEntry(root_nidx, p_tree->GetDepth(root_nidx), splits.front(), 0));
  }

  void UpdatePosition(const ExpandEntry& candidate, RegTree* p_tree) {
    auto nidx = candidate.nid;
    auto left_nidx = (*p_tree)[nidx].LeftChild();
    auto right_nidx = (*p_tree)[nidx].RightChild();

    // convert floating-point split_pt into corresponding bin_id
    // split_cond = -1 indicates that split_pt is less than all known cut points
    auto split_gidx = -1;
    auto fidx = candidate.split.findex;
    auto default_dir_left = candidate.split.dir == kLeftDir;
    auto fidx_begin = hmat_.row_ptr[fidx];
    auto fidx_end = hmat_.row_ptr[fidx + 1];
    for (auto i = fidx_begin; i < fidx_end; ++i) {
      if (candidate.split.fvalue == hmat_.cut[i]) {
        split_gidx = static_cast<int32_t>(i);
      }
    }

    auto is_dense = info_->num_nonzero_ == info_->num_row_ * info_->num_col_;

    omp_set_num_threads(shards_.size());
#pragma omp parallel
    {
      auto cpu_thread_id = omp_get_thread_num();
      shards_[cpu_thread_id]->UpdatePosition(nidx, left_nidx, right_nidx, fidx,
                                            split_gidx, default_dir_left,
                                            is_dense, fidx_begin, fidx_end);
    }
  }

  void ApplySplit(const ExpandEntry& candidate, RegTree* p_tree) {
    // Add new leaves
    RegTree& tree = *p_tree;
    tree.AddChilds(candidate.nid);
    auto& parent = tree[candidate.nid];
    parent.SetSplit(candidate.split.findex, candidate.split.fvalue,
                     candidate.split.dir == kLeftDir);
    tree.Stat(candidate.nid).loss_chg = candidate.split.loss_chg;

    // Set up child constraints
    node_value_constraints_.resize(tree.GetNodes().size());
    GradStats left_stats(param_);
    left_stats.Add(candidate.split.left_sum);
    GradStats right_stats(param_);
    right_stats.Add(candidate.split.right_sum);
    node_value_constraints_[candidate.nid].SetChild(
        param_, parent.SplitIndex(), left_stats, right_stats,
        &node_value_constraints_[parent.LeftChild()],
        &node_value_constraints_[parent.RightChild()]);

    // Configure left child
    auto left_weight =
        node_value_constraints_[parent.LeftChild()].CalcWeight(param_, left_stats);
    tree[parent.LeftChild()].SetLeaf(left_weight * param_.learning_rate, 0);
    tree.Stat(parent.LeftChild()).base_weight = left_weight;
    tree.Stat(parent.LeftChild()).sum_hess = candidate.split.left_sum.GetHess();

    // Configure right child
    auto right_weight =
        node_value_constraints_[parent.RightChild()].CalcWeight(param_, right_stats);
    tree[parent.RightChild()].SetLeaf(right_weight * param_.learning_rate, 0);
    tree.Stat(parent.RightChild()).base_weight = right_weight;
    tree.Stat(parent.RightChild()).sum_hess = candidate.split.right_sum.GetHess();
    // Store sum gradients
    for (auto& shard : shards_) {
      shard->node_sum_gradients[parent.LeftChild()] = candidate.split.left_sum;
      shard->node_sum_gradients[parent.RightChild()] = candidate.split.right_sum;
    }
    this->UpdatePosition(candidate, p_tree);
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* p_fmat,
                  RegTree* p_tree) {
    // Temporarily store number of threads so we can change it back later
    int nthread = omp_get_max_threads();

    auto& tree = *p_tree;

    monitor_.Start("InitData", device_list_);
    this->InitData(gpair, p_fmat, *p_tree);
    monitor_.Stop("InitData", device_list_);
    monitor_.Start("InitRoot", device_list_);
    this->InitRoot(p_tree);
    monitor_.Stop("InitRoot", device_list_);

    auto timestamp = qexpand_->size();
    auto num_leaves = 1;

    while (!qexpand_->empty()) {
      auto candidate = qexpand_->top();
      qexpand_->pop();
      if (!candidate.IsValid(param_, num_leaves)) continue;
      // std::cout << candidate;
      monitor_.Start("ApplySplit", device_list_);
      this->ApplySplit(candidate, p_tree);
      monitor_.Stop("ApplySplit", device_list_);
      num_leaves++;

      auto left_child_nidx = tree[candidate.nid].LeftChild();
      auto right_child_nidx = tree[candidate.nid].RightChild();

      // Only create child entries if needed
      if (ExpandEntry::ChildIsValid(param_, tree.GetDepth(left_child_nidx),
                                    num_leaves)) {
        monitor_.Start("BuildHist", device_list_);
        this->BuildHistLeftRight(candidate.nid, left_child_nidx,
                                 right_child_nidx);
        monitor_.Stop("BuildHist", device_list_);

        monitor_.Start("EvaluateSplits", device_list_);
        auto splits =
            this->EvaluateSplits({left_child_nidx, right_child_nidx}, p_tree);
        qexpand_->push(ExpandEntry(left_child_nidx,
                                   tree.GetDepth(left_child_nidx), splits[0],
                                   timestamp++));
        qexpand_->push(ExpandEntry(right_child_nidx,
                                   tree.GetDepth(right_child_nidx), splits[1],
                                   timestamp++));
        monitor_.Stop("EvaluateSplits", device_list_);
      }
    }
    // Reset omp num threads
    omp_set_num_threads(nthread);
  }

  bool UpdatePredictionCache(
      const DMatrix* data, HostDeviceVector<bst_float>* p_out_preds) override {
    monitor_.Start("UpdatePredictionCache", device_list_);
    if (shards_.empty() || p_last_fmat_ == nullptr || p_last_fmat_ != data)
      return false;

    bst_float* out_preds_d = p_out_preds->DevicePointer(param_.gpu_id);

#pragma omp parallel for schedule(static, 1)
    for (int shard = 0; shard < shards_.size(); ++shard) {
      shards_[shard]->UpdatePredictionCache(out_preds_d);
    }
    monitor_.Stop("UpdatePredictionCache", device_list_);
    return true;
  }

  struct ExpandEntry {
    int nid;
    int depth;
    DeviceSplitCandidate split;
    uint64_t timestamp;
    ExpandEntry(int nid, int depth, const DeviceSplitCandidate& split,
                uint64_t timestamp)
        : nid(nid), depth(depth), split(split), timestamp(timestamp) {}
    bool IsValid(const TrainParam& param, int num_leaves) const {
      if (split.loss_chg <= kRtEps) return false;
      if (split.left_sum.GetHess() == 0 || split.right_sum.GetHess() == 0)
        return false;
      if (param.max_depth > 0 && depth == param.max_depth) return false;
      if (param.max_leaves > 0 && num_leaves == param.max_leaves) return false;
      return true;
    }

    static bool ChildIsValid(const TrainParam& param, int depth,
                             int num_leaves) {
      if (param.max_depth > 0 && depth == param.max_depth) return false;
      if (param.max_leaves > 0 && num_leaves == param.max_leaves) return false;
      return true;
    }

    friend std::ostream& operator<<(std::ostream& os, const ExpandEntry& e) {
      os << "ExpandEntry: \n";
      os << "nidx: " << e.nid << "\n";
      os << "depth: " << e.depth << "\n";
      os << "loss: " << e.split.loss_chg << "\n";
      os << "left_sum: " << e.split.left_sum << "\n";
      os << "right_sum: " << e.split.right_sum << "\n";
      return os;
    }
  };

  inline static bool DepthWise(ExpandEntry lhs, ExpandEntry rhs) {
    if (lhs.depth == rhs.depth) {
      return lhs.timestamp > rhs.timestamp;  // favor small timestamp
    } else {
      return lhs.depth > rhs.depth;  // favor small depth
    }
  }
  inline static bool LossGuide(ExpandEntry lhs, ExpandEntry rhs) {
    if (lhs.split.loss_chg == rhs.split.loss_chg) {
      return lhs.timestamp > rhs.timestamp;  // favor small timestamp
    } else {
      return lhs.split.loss_chg < rhs.split.loss_chg;  // favor large loss_chg
    }
  }
  TrainParam param_;
  common::HistCutMatrix hmat_;
  common::GHistIndexMatrix gmat_;
  MetaInfo* info_;
  bool initialised_;
  int n_devices_;
  int n_bins_;

  std::vector<std::unique_ptr<DeviceShard>> shards_;
  ColumnSampler column_sampler_;
  typedef std::priority_queue<ExpandEntry, std::vector<ExpandEntry>,
                              std::function<bool(ExpandEntry, ExpandEntry)>>
      ExpandQueue;
  std::unique_ptr<ExpandQueue> qexpand_;
  common::Monitor monitor_;
  dh::AllReducer reducer_;
  std::vector<ValueConstraint> node_value_constraints_;
  std::vector<int> device_list_;

  DMatrix* p_last_fmat_;
};

XGBOOST_REGISTER_TREE_UPDATER(GPUHistMaker, "grow_gpu_hist")
    .describe("Grow tree with GPU.")
    .set_body([]() { return new GPUHistMaker(); });
}  // namespace tree
}  // namespace xgboost
