#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2019 XGBoost contributors
 */
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <xgboost/tree_updater.h>
#include <algorithm>
#include <cmath>
#include <memory>
#include <limits>
#include <queue>
#include <utility>
#include <vector>

#include "xgboost/host_device_vector.h"
#include "xgboost/parameter.h"
#include "xgboost/span.h"
#include "xgboost/json.h"

#include "../common/common.h"
#include "../common/compressed_iterator.h"
#include "../common/device_helpers.cuh"
#include "../common/hist_util.h"
#include "../common/timer.h"
#include "../data/ellpack_page.cuh"
#include "param.h"
#include "updater_gpu_common.cuh"
#include "constraints.cuh"
#include "gpu_hist/row_partitioner.cuh"

namespace xgboost {
namespace tree {

#if !defined(GTEST_TEST)
DMLC_REGISTRY_FILE_TAG(updater_gpu_hist);
#endif  // !defined(GTEST_TEST)

// training parameters specific to this algorithm
struct GPUHistMakerTrainParam
    : public XGBoostParameter<GPUHistMakerTrainParam> {
  bool single_precision_histogram;
  // number of rows in a single GPU batch
  int gpu_batch_nrows;
  bool debug_synchronize;
  // declare parameters
  DMLC_DECLARE_PARAMETER(GPUHistMakerTrainParam) {
    DMLC_DECLARE_FIELD(single_precision_histogram).set_default(false).describe(
        "Use single precision to build histograms.");
    DMLC_DECLARE_FIELD(gpu_batch_nrows)
        .set_lower_bound(-1)
        .set_default(0)
        .describe("Number of rows in a GPU batch, used for finding quantiles on GPU; "
                  "-1 to use all rows assignted to a GPU, and 0 to auto-deduce");
    DMLC_DECLARE_FIELD(debug_synchronize).set_default(false).describe(
        "Check if all distributed tree are identical after tree construction.");
  }
};
#if !defined(GTEST_TEST)
DMLC_REGISTER_PARAMETER(GPUHistMakerTrainParam);
#endif  // !defined(GTEST_TEST)

struct ExpandEntry {
  int nid;
  int depth;
  DeviceSplitCandidate split;
  uint64_t timestamp;
  ExpandEntry() = default;
  ExpandEntry(int nid, int depth, DeviceSplitCandidate split,
              uint64_t timestamp)
      : nid(nid), depth(depth), split(std::move(split)), timestamp(timestamp) {}
  bool IsValid(const TrainParam& param, int num_leaves) const {
    if (split.loss_chg <= kRtEps) return false;
    if (split.left_sum.GetHess() == 0 || split.right_sum.GetHess() == 0) {
      return false;
    }
    if (split.loss_chg < param.min_split_loss) { return false; }
    if (param.max_depth > 0 && depth == param.max_depth) {return false; }
    if (param.max_leaves > 0 && num_leaves == param.max_leaves) { return false; }
    return true;
  }

  static bool ChildIsValid(const TrainParam& param, int depth, int num_leaves) {
    if (param.max_depth > 0 && depth >= param.max_depth) return false;
    if (param.max_leaves > 0 && num_leaves >= param.max_leaves) return false;
    return true;
  }

  friend std::ostream& operator<<(std::ostream& os, const ExpandEntry& e) {
    os << "ExpandEntry: \n";
    os << "nidx: " << e.nid << "\n";
    os << "depth: " << e.depth << "\n";
    os << "loss: " << e.split.loss_chg << "\n";
    os << "left_sum: " << e.split.left_sum << "\n";
    os << "right_sum: " << e.split.right_sum << "\n";
    return os;
  }
};

inline static bool DepthWise(const ExpandEntry& lhs, const ExpandEntry& rhs) {
  if (lhs.depth == rhs.depth) {
    return lhs.timestamp > rhs.timestamp;  // favor small timestamp
  } else {
    return lhs.depth > rhs.depth;  // favor small depth
  }
}
inline static bool LossGuide(const ExpandEntry& lhs, const ExpandEntry& rhs) {
  if (lhs.split.loss_chg == rhs.split.loss_chg) {
    return lhs.timestamp > rhs.timestamp;  // favor small timestamp
  } else {
    return lhs.split.loss_chg < rhs.split.loss_chg;  // favor large loss_chg
  }
}

// With constraints
template <typename GradientPairT>
XGBOOST_DEVICE float inline LossChangeMissing(
    const GradientPairT& scan, const GradientPairT& missing, const GradientPairT& parent_sum,
    const float& parent_gain, const GPUTrainingParam& param, int constraint,
    const ValueConstraint& value_constraint,
    bool& missing_left_out) {  // NOLINT
  float missing_left_gain = value_constraint.CalcSplitGain(
      param, constraint, GradStats(scan + missing),
      GradStats(parent_sum - (scan + missing)));
  float missing_right_gain = value_constraint.CalcSplitGain(
      param, constraint, GradStats(scan), GradStats(parent_sum - scan));

  if (missing_left_gain >= missing_right_gain) {
    missing_left_out = true;
    return missing_left_gain - parent_gain;
  } else {
    missing_left_out = false;
    return missing_right_gain - parent_gain;
  }
}

/*!
 * \brief
 *
 * \tparam ReduceT     BlockReduce Type.
 * \tparam TempStorage Cub Shared memory
 *
 * \param begin
 * \param end
 * \param temp_storage Shared memory for intermediate result.
 */
template <int BLOCK_THREADS, typename ReduceT, typename TempStorageT, typename GradientSumT>
__device__ GradientSumT ReduceFeature(common::Span<const GradientSumT> feature_histogram,
                                      TempStorageT* temp_storage) {
  __shared__ hipcub::Uninitialized<GradientSumT> uninitialized_sum;
  GradientSumT& shared_sum = uninitialized_sum.Alias();

  GradientSumT local_sum = GradientSumT();
  // For loop sums features into one block size
  auto begin = feature_histogram.data();
  auto end = begin + feature_histogram.size();
  for (auto itr = begin; itr < end; itr += BLOCK_THREADS) {
    bool thread_active = itr + threadIdx.x < end;
    // Scan histogram
    GradientSumT bin = thread_active ? *(itr + threadIdx.x) : GradientSumT();
    local_sum += bin;
  }
  local_sum = ReduceT(temp_storage->sum_reduce).Reduce(local_sum, hipcub::Sum());
  // Reduction result is stored in thread 0.
  if (threadIdx.x == 0) {
    shared_sum = local_sum;
  }
  __syncthreads();
  return shared_sum;
}

/*! \brief Find the thread with best gain. */
template <int BLOCK_THREADS, typename ReduceT, typename ScanT,
          typename MaxReduceT, typename TempStorageT, typename GradientSumT>
__device__ void EvaluateFeature(
    int fidx, common::Span<const GradientSumT> node_histogram,
    const xgboost::EllpackMatrix& matrix,
    DeviceSplitCandidate* best_split,  // shared memory storing best split
    const DeviceNodeStats& node, const GPUTrainingParam& param,
    TempStorageT* temp_storage,  // temp memory for cub operations
    int constraint,              // monotonic_constraints
    const ValueConstraint& value_constraint) {
  // Use pointer from cut to indicate begin and end of bins for each feature.
  uint32_t gidx_begin = matrix.info.feature_segments[fidx];  // begining bin
  uint32_t gidx_end = matrix.info.feature_segments[fidx + 1];  // end bin for i^th feature

  // Sum histogram bins for current feature
  GradientSumT const feature_sum = ReduceFeature<BLOCK_THREADS, ReduceT>(
      node_histogram.subspan(gidx_begin, gidx_end - gidx_begin), temp_storage);

  GradientSumT const parent_sum = GradientSumT(node.sum_gradients);
  GradientSumT const missing = parent_sum - feature_sum;
  float const null_gain = -std::numeric_limits<bst_float>::infinity();

  SumCallbackOp<GradientSumT> prefix_op =
      SumCallbackOp<GradientSumT>();
  for (int scan_begin = gidx_begin; scan_begin < gidx_end;
       scan_begin += BLOCK_THREADS) {
    bool thread_active = (scan_begin + threadIdx.x) < gidx_end;

    // Gradient value for current bin.
    GradientSumT bin =
        thread_active ? node_histogram[scan_begin + threadIdx.x] : GradientSumT();
    ScanT(temp_storage->scan).ExclusiveScan(bin, bin, hipcub::Sum(), prefix_op);

    // Whether the gradient of missing values is put to the left side.
    bool missing_left = true;
    float gain = null_gain;
    if (thread_active) {
      gain = LossChangeMissing(bin, missing, parent_sum, node.root_gain, param,
                               constraint, value_constraint, missing_left);
    }

    __syncthreads();

    // Find thread with best gain
    hipcub::KeyValuePair<int, float> tuple(threadIdx.x, gain);
    hipcub::KeyValuePair<int, float> best =
        MaxReduceT(temp_storage->max_reduce).Reduce(tuple, hipcub::ArgMax());

    __shared__ hipcub::KeyValuePair<int, float> block_max;
    if (threadIdx.x == 0) {
      block_max = best;
    }

    __syncthreads();

    // Best thread updates split
    if (threadIdx.x == block_max.key) {
      int split_gidx = (scan_begin + threadIdx.x) - 1;
      float fvalue;
      if (split_gidx < static_cast<int>(gidx_begin)) {
        fvalue =  matrix.info.min_fvalue[fidx];
      } else {
        fvalue = matrix.info.gidx_fvalue_map[split_gidx];
      }
      GradientSumT left = missing_left ? bin + missing : bin;
      GradientSumT right = parent_sum - left;
      best_split->Update(gain, missing_left ? kLeftDir : kRightDir, fvalue,
                         fidx, GradientPair(left), GradientPair(right), param);
    }
    __syncthreads();
  }
}

template <int BLOCK_THREADS, typename GradientSumT>
__global__ void EvaluateSplitKernel(
    common::Span<const GradientSumT> node_histogram,  // histogram for gradients
    common::Span<const bst_feature_t> feature_set,    // Selected features
    DeviceNodeStats node,
    xgboost::EllpackMatrix matrix,
    GPUTrainingParam gpu_param,
    common::Span<DeviceSplitCandidate> split_candidates,  // resulting split
    ValueConstraint value_constraint,
    common::Span<int> d_monotonic_constraints) {
  // KeyValuePair here used as threadIdx.x -> gain_value
  using ArgMaxT = hipcub::KeyValuePair<int, float>;
  using BlockScanT =
      hipcub::BlockScan<GradientSumT, BLOCK_THREADS, hipcub::BLOCK_SCAN_WARP_SCANS>;
  using MaxReduceT = hipcub::BlockReduce<ArgMaxT, BLOCK_THREADS>;

  using SumReduceT = hipcub::BlockReduce<GradientSumT, BLOCK_THREADS>;

  union TempStorage {
    typename BlockScanT::TempStorage scan;
    typename MaxReduceT::TempStorage max_reduce;
    typename SumReduceT::TempStorage sum_reduce;
  };

  // Aligned && shared storage for best_split
  __shared__ hipcub::Uninitialized<DeviceSplitCandidate> uninitialized_split;
  DeviceSplitCandidate& best_split = uninitialized_split.Alias();
  __shared__ TempStorage temp_storage;

  if (threadIdx.x == 0) {
    best_split = DeviceSplitCandidate();
  }

  __syncthreads();

  // One block for each feature. Features are sampled, so fidx != blockIdx.x
  int fidx = feature_set[blockIdx.x];

  int constraint = d_monotonic_constraints[fidx];
  EvaluateFeature<BLOCK_THREADS, SumReduceT, BlockScanT, MaxReduceT>(
      fidx, node_histogram, matrix, &best_split, node, gpu_param, &temp_storage,
      constraint, value_constraint);

  __syncthreads();

  if (threadIdx.x == 0) {
    // Record best loss for each feature
    split_candidates[blockIdx.x] = best_split;
  }
}

/**
 * \struct  DeviceHistogram
 *
 * \summary Data storage for node histograms on device. Automatically expands.
 *
 * \tparam GradientSumT      histogram entry type.
 * \tparam kStopGrowingSize  Do not grow beyond this size
 *
 * \author  Rory
 * \date    28/07/2018
 */
template <typename GradientSumT, size_t kStopGrowingSize = 1 << 26>
class DeviceHistogram {
 private:
  /*! \brief Map nidx to starting index of its histogram. */
  std::map<int, size_t> nidx_map_;
  dh::device_vector<typename GradientSumT::ValueT> data_;
  int n_bins_;
  int device_id_;
  static constexpr size_t kNumItemsInGradientSum =
      sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT);
  static_assert(kNumItemsInGradientSum == 2,
                "Number of items in gradient type should be 2.");

 public:
  void Init(int device_id, int n_bins) {
    this->n_bins_ = n_bins;
    this->device_id_ = device_id;
  }

  void Reset() {
    dh::safe_cuda(hipMemsetAsync(
        data_.data().get(), 0,
        data_.size() * sizeof(typename decltype(data_)::value_type)));
    nidx_map_.clear();
  }
  bool HistogramExists(int nidx) const {
    return nidx_map_.find(nidx) != nidx_map_.cend();
  }
  size_t HistogramSize() const {
    return n_bins_ * kNumItemsInGradientSum;
  }

  dh::device_vector<typename GradientSumT::ValueT>& Data() {
    return data_;
  }

  void AllocateHistogram(int nidx) {
    if (HistogramExists(nidx)) return;
    // Number of items currently used in data
    const size_t used_size = nidx_map_.size() * HistogramSize();
    const size_t new_used_size = used_size + HistogramSize();
    dh::safe_cuda(hipSetDevice(device_id_));
    if (data_.size() >= kStopGrowingSize) {
      // Recycle histogram memory
      if (new_used_size <= data_.size()) {
        // no need to remove old node, just insert the new one.
        nidx_map_[nidx] = used_size;
        // memset histogram size in bytes
        dh::safe_cuda(hipMemsetAsync(data_.data().get() + used_size, 0,
                                      n_bins_ * sizeof(GradientSumT)));
      } else {
        std::pair<int, size_t> old_entry = *nidx_map_.begin();
        nidx_map_.erase(old_entry.first);
        dh::safe_cuda(hipMemsetAsync(data_.data().get() + old_entry.second, 0,
                                      n_bins_ * sizeof(GradientSumT)));
        nidx_map_[nidx] = old_entry.second;
      }
    } else {
      // Append new node histogram
      nidx_map_[nidx] = used_size;
      size_t new_required_memory = std::max(data_.size() * 2, HistogramSize());
      if (data_.size() < new_required_memory) {
        data_.resize(new_required_memory);
      }
    }
  }

  /**
   * \summary   Return pointer to histogram memory for a given node.
   * \param nidx    Tree node index.
   * \return    hist pointer.
   */
  common::Span<GradientSumT> GetNodeHistogram(int nidx) {
    CHECK(this->HistogramExists(nidx));
    auto ptr = data_.data().get() + nidx_map_[nidx];
    return common::Span<GradientSumT>(
        reinterpret_cast<GradientSumT*>(ptr), n_bins_);
  }
};

struct CalcWeightTrainParam {
  float min_child_weight;
  float reg_alpha;
  float reg_lambda;
  float max_delta_step;
  float learning_rate;
  XGBOOST_DEVICE explicit CalcWeightTrainParam(const TrainParam& p)
      : min_child_weight(p.min_child_weight),
        reg_alpha(p.reg_alpha),
        reg_lambda(p.reg_lambda),
        max_delta_step(p.max_delta_step),
        learning_rate(p.learning_rate) {}
};

template <typename GradientSumT>
__global__ void SharedMemHistKernel(xgboost::EllpackMatrix matrix,
                                    common::Span<const RowPartitioner::RowIndexT> d_ridx,
                                    GradientSumT* d_node_hist,
                                    const GradientPair* d_gpair, size_t n_elements,
                                    bool use_shared_memory_histograms) {
  extern __shared__ char smem[];
  GradientSumT* smem_arr = reinterpret_cast<GradientSumT*>(smem);  // NOLINT
  if (use_shared_memory_histograms) {
    dh::BlockFill(smem_arr, matrix.info.n_bins, GradientSumT());
    __syncthreads();
  }
  for (auto idx : dh::GridStrideRange(static_cast<size_t>(0), n_elements)) {
    int ridx = d_ridx[idx / matrix.info.row_stride];
    if (!matrix.IsInRange(ridx)) {
      continue;
    }
    int gidx = matrix.gidx_iter[(ridx - matrix.base_rowid) * matrix.info.row_stride
        + idx % matrix.info.row_stride];
    if (gidx != matrix.info.n_bins) {
      // If we are not using shared memory, accumulate the values directly into
      // global memory
      GradientSumT* atomic_add_ptr =
          use_shared_memory_histograms ? smem_arr : d_node_hist;
      dh::AtomicAddGpair(atomic_add_ptr + gidx, d_gpair[ridx]);
    }
  }

  if (use_shared_memory_histograms) {
    // Write shared memory back to global memory
    __syncthreads();
    for (auto i : dh::BlockStrideRange(static_cast<size_t>(0), matrix.info.n_bins)) {
      dh::AtomicAddGpair(d_node_hist + i, smem_arr[i]);
    }
  }
}

// Manage memory for a single GPU
template <typename GradientSumT>
struct GPUHistMakerDevice {
  int device_id;
  EllpackPageImpl* page;
  BatchParam batch_param;

  dh::BulkAllocator ba;

  std::unique_ptr<RowPartitioner> row_partitioner;
  DeviceHistogram<GradientSumT> hist{};

  /*! \brief Gradient pair for each row. */
  common::Span<GradientPair> gpair;

  common::Span<int> monotone_constraints;
  common::Span<bst_float> prediction_cache;

  /*! \brief Sum gradient for each node. */
  std::vector<GradientPair> node_sum_gradients;
  common::Span<GradientPair> node_sum_gradients_d;
  bst_uint n_rows;

  TrainParam param;
  bool prediction_cache_initialised;
  bool use_shared_memory_histograms {false};

  dh::CubMemory temp_memory;
  dh::PinnedMemory pinned_memory;

  std::vector<hipStream_t> streams{};

  common::Monitor monitor;
  std::vector<ValueConstraint> node_value_constraints;
  common::ColumnSampler column_sampler;
  FeatureInteractionConstraint interaction_constraints;

  using ExpandQueue =
      std::priority_queue<ExpandEntry, std::vector<ExpandEntry>,
                          std::function<bool(ExpandEntry, ExpandEntry)>>;
  std::unique_ptr<ExpandQueue> qexpand;

  GPUHistMakerDevice(int _device_id,
                     EllpackPageImpl* _page,
                     bst_uint _n_rows,
                     TrainParam _param,
                     uint32_t column_sampler_seed,
                     uint32_t n_features,
                     BatchParam _batch_param)
      : device_id(_device_id),
        page(_page),
        n_rows(_n_rows),
        param(std::move(_param)),
        prediction_cache_initialised(false),
        column_sampler(column_sampler_seed),
        interaction_constraints(param, n_features),
        batch_param(_batch_param) {
    monitor.Init(std::string("GPUHistMakerDevice") + std::to_string(device_id));
  }

  void InitHistogram();

  ~GPUHistMakerDevice() {  // NOLINT
    dh::safe_cuda(hipSetDevice(device_id));
    for (auto& stream : streams) {
      dh::safe_cuda(hipStreamDestroy(stream));
    }
  }

  // Get vector of at least n initialised streams
  std::vector<hipStream_t>& GetStreams(int n) {
    if (n > streams.size()) {
      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamDestroy(stream));
      }

      streams.clear();
      streams.resize(n);

      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamCreate(&stream));
      }
    }

    return streams;
  }

  // Reset values for each update iteration
  // Note that the column sampler must be passed by value because it is not
  // thread safe
  void Reset(HostDeviceVector<GradientPair>* dh_gpair, int64_t num_columns) {
    if (param.grow_policy == TrainParam::kLossGuide) {
      qexpand.reset(new ExpandQueue(LossGuide));
    } else {
      qexpand.reset(new ExpandQueue(DepthWise));
    }
    this->column_sampler.Init(num_columns, param.colsample_bynode,
      param.colsample_bylevel, param.colsample_bytree);
    dh::safe_cuda(hipSetDevice(device_id));
    this->interaction_constraints.Reset();
    std::fill(node_sum_gradients.begin(), node_sum_gradients.end(),
              GradientPair());
    row_partitioner.reset();  // Release the device memory first before reallocating
    row_partitioner.reset(new RowPartitioner(device_id, n_rows));

    dh::safe_cuda(hipMemcpyAsync(
        gpair.data(), dh_gpair->ConstDevicePointer(),
        gpair.size() * sizeof(GradientPair), hipMemcpyHostToHost));
    SubsampleGradientPair(device_id, gpair, param.subsample);
    hist.Reset();
  }

  std::vector<DeviceSplitCandidate> EvaluateSplits(
      std::vector<int> nidxs, const RegTree& tree,
      size_t num_columns) {
    dh::safe_cuda(hipSetDevice(device_id));
    auto result_all = pinned_memory.GetSpan<DeviceSplitCandidate>(nidxs.size());

    // Work out cub temporary memory requirement
    GPUTrainingParam gpu_param(param);
    DeviceSplitCandidateReduceOp op(gpu_param);
    size_t temp_storage_bytes = 0;
    DeviceSplitCandidate*dummy = nullptr;
    hipcub::DeviceReduce::Reduce(
        nullptr, temp_storage_bytes, dummy,
        dummy, num_columns, op,
        DeviceSplitCandidate());
    // size in terms of DeviceSplitCandidate
    size_t cub_memory_size =
      std::ceil(static_cast<double>(temp_storage_bytes) /
        sizeof(DeviceSplitCandidate));

    // Allocate enough temporary memory
    // Result for each nidx
    // + intermediate result for each column
    // + cub reduce memory
    auto temp_span = temp_memory.GetSpan<DeviceSplitCandidate>(
        nidxs.size() + nidxs.size() * num_columns +cub_memory_size*nidxs.size());
    auto d_result_all = temp_span.subspan(0, nidxs.size());
    auto d_split_candidates_all =
        temp_span.subspan(d_result_all.size(), nidxs.size() * num_columns);
    auto d_cub_memory_all =
        temp_span.subspan(d_result_all.size() + d_split_candidates_all.size(),
                          cub_memory_size * nidxs.size());

    auto& streams = this->GetStreams(nidxs.size());
    for (auto i = 0ull; i < nidxs.size(); i++) {
      auto nidx = nidxs[i];
      auto p_feature_set = column_sampler.GetFeatureSet(tree.GetDepth(nidx));
      p_feature_set->SetDevice(device_id);
      common::Span<bst_feature_t> d_sampled_features = p_feature_set->DeviceSpan();
      common::Span<bst_feature_t> d_feature_set =
          interaction_constraints.Query(d_sampled_features, nidx);
      auto d_split_candidates =
          d_split_candidates_all.subspan(i * num_columns, d_feature_set.size());

      DeviceNodeStats node(node_sum_gradients[nidx], nidx, param);

      auto d_result = d_result_all.subspan(i, 1);
      if (d_feature_set.empty()) {
        // Acting as a device side constructor for DeviceSplitCandidate.
        // DeviceSplitCandidate::IsValid is false so that ApplySplit can reject this
        // candidate.
        auto worst_candidate = DeviceSplitCandidate();
        dh::safe_cuda(hipMemcpyAsync(d_result.data(), &worst_candidate,
                                      sizeof(DeviceSplitCandidate),
                                      hipMemcpyHostToDevice));
        continue;
      }

      // One block for each feature
      uint32_t constexpr kBlockThreads = 256;
      dh::LaunchKernel {uint32_t(d_feature_set.size()), kBlockThreads, 0, streams[i]} (
          EvaluateSplitKernel<kBlockThreads, GradientSumT>,
          hist.GetNodeHistogram(nidx), d_feature_set, node, page->matrix,
          gpu_param, d_split_candidates, node_value_constraints[nidx],
          monotone_constraints);

      // Reduce over features to find best feature
      auto d_cub_memory =
          d_cub_memory_all.subspan(i * cub_memory_size, cub_memory_size);
      size_t cub_bytes = d_cub_memory.size() * sizeof(DeviceSplitCandidate);
      hipcub::DeviceReduce::Reduce(reinterpret_cast<void*>(d_cub_memory.data()),
                                cub_bytes, d_split_candidates.data(),
                                d_result.data(), d_split_candidates.size(), op,
                                DeviceSplitCandidate(), streams[i]);
    }

    dh::safe_cuda(hipMemcpy(result_all.data(), d_result_all.data(),
                             sizeof(DeviceSplitCandidate) * d_result_all.size(),
                             hipMemcpyDeviceToHost));
    return std::vector<DeviceSplitCandidate>(result_all.begin(), result_all.end());
  }

  // Build gradient histograms for a given node across all the batches in the DMatrix.
  void BuildHistBatches(int nidx, DMatrix* p_fmat) {
    for (auto& batch : p_fmat->GetBatches<EllpackPage>(batch_param)) {
      page = batch.Impl();
      BuildHist(nidx);
    }
  }

  void BuildHist(int nidx) {
    hist.AllocateHistogram(nidx);
    auto d_node_hist = hist.GetNodeHistogram(nidx);
    auto d_ridx = row_partitioner->GetRows(nidx);
    auto d_gpair = gpair.data();

    auto n_elements = d_ridx.size() * page->matrix.info.row_stride;

    const size_t smem_size =
        use_shared_memory_histograms
            ? sizeof(GradientSumT) * page->matrix.info.n_bins
            : 0;
    uint32_t items_per_thread = 8;
    uint32_t block_threads = 256;
    auto grid_size = static_cast<uint32_t>(
        common::DivRoundUp(n_elements, items_per_thread * block_threads));
    dh::LaunchKernel {grid_size, block_threads, smem_size} (
        SharedMemHistKernel<GradientSumT>,
        page->matrix, d_ridx, d_node_hist.data(), d_gpair, n_elements,
        use_shared_memory_histograms);
  }

  void SubtractionTrick(int nidx_parent, int nidx_histogram,
                        int nidx_subtraction) {
    auto d_node_hist_parent = hist.GetNodeHistogram(nidx_parent);
    auto d_node_hist_histogram = hist.GetNodeHistogram(nidx_histogram);
    auto d_node_hist_subtraction = hist.GetNodeHistogram(nidx_subtraction);

    dh::LaunchN(device_id, page->matrix.info.n_bins, [=] __device__(size_t idx) {
      d_node_hist_subtraction[idx] =
          d_node_hist_parent[idx] - d_node_hist_histogram[idx];
    });
  }

  bool CanDoSubtractionTrick(int nidx_parent, int nidx_histogram,
                             int nidx_subtraction) {
    // Make sure histograms are already allocated
    hist.AllocateHistogram(nidx_subtraction);
    return hist.HistogramExists(nidx_histogram) &&
           hist.HistogramExists(nidx_parent);
  }

  void UpdatePosition(int nidx, RegTree::Node split_node) {
    auto d_matrix = page->matrix;

    row_partitioner->UpdatePosition(
        nidx, split_node.LeftChild(), split_node.RightChild(),
        [=] __device__(size_t ridx) {
          if (!d_matrix.IsInRange(ridx)) {
            return RowPartitioner::kIgnoredTreePosition;
          }
          // given a row index, returns the node id it belongs to
          bst_float cut_value =
              d_matrix.GetElement(ridx, split_node.SplitIndex());
          // Missing value
          int new_position = 0;
          if (isnan(cut_value)) {
            new_position = split_node.DefaultChild();
          } else {
            if (cut_value <= split_node.SplitCond()) {
              new_position = split_node.LeftChild();
            } else {
              new_position = split_node.RightChild();
            }
          }
          return new_position;
        });
  }

  // After tree update is finished, update the position of all training
  // instances to their final leaf. This information is used later to update the
  // prediction cache
  void FinalisePosition(RegTree* p_tree, DMatrix* p_fmat) {
    const auto d_nodes =
        temp_memory.GetSpan<RegTree::Node>(p_tree->GetNodes().size());
    dh::safe_cuda(hipMemcpy(d_nodes.data(), p_tree->GetNodes().data(),
                             d_nodes.size() * sizeof(RegTree::Node),
                             hipMemcpyHostToDevice));

    for (auto& batch : p_fmat->GetBatches<EllpackPage>(batch_param)) {
      page = batch.Impl();
      auto d_matrix = page->matrix;
      row_partitioner->FinalisePosition(
          [=] __device__(size_t row_id, int position) {
            if (!d_matrix.IsInRange(row_id)) {
              return RowPartitioner::kIgnoredTreePosition;
            }
            auto node = d_nodes[position];

            while (!node.IsLeaf()) {
              bst_float element = d_matrix.GetElement(row_id, node.SplitIndex());
              // Missing value
              if (isnan(element)) {
                position = node.DefaultChild();
              } else {
                if (element <= node.SplitCond()) {
                  position = node.LeftChild();
                } else {
                  position = node.RightChild();
                }
              }
              node = d_nodes[position];
            }
            return position;
          });
    }
  }

  void UpdatePredictionCache(bst_float* out_preds_d) {
    dh::safe_cuda(hipSetDevice(device_id));
    if (!prediction_cache_initialised) {
      dh::safe_cuda(hipMemcpyAsync(prediction_cache.data(), out_preds_d,
                                    prediction_cache.size() * sizeof(bst_float),
                                    hipMemcpyDefault));
    }
    prediction_cache_initialised = true;

    CalcWeightTrainParam param_d(param);

    dh::safe_cuda(
        hipMemcpyAsync(node_sum_gradients_d.data(), node_sum_gradients.data(),
                        sizeof(GradientPair) * node_sum_gradients.size(),
                        hipMemcpyHostToDevice));
    auto d_position = row_partitioner->GetPosition();
    auto d_ridx = row_partitioner->GetRows();
    auto d_node_sum_gradients = node_sum_gradients_d.data();
    auto d_prediction_cache = prediction_cache.data();

    dh::LaunchN(
        device_id, prediction_cache.size(), [=] __device__(int local_idx) {
          int pos = d_position[local_idx];
          bst_float weight = CalcWeight(param_d, d_node_sum_gradients[pos]);
          d_prediction_cache[d_ridx[local_idx]] +=
              weight * param_d.learning_rate;
        });

    dh::safe_cuda(hipMemcpy(
        out_preds_d, prediction_cache.data(),
        prediction_cache.size() * sizeof(bst_float), hipMemcpyDefault));
    row_partitioner.reset();
  }

  void AllReduceHist(int nidx, dh::AllReducer* reducer) {
    monitor.StartCuda("AllReduce");
    auto d_node_hist = hist.GetNodeHistogram(nidx).data();
    reducer->AllReduceSum(
        reinterpret_cast<typename GradientSumT::ValueT*>(d_node_hist),
        reinterpret_cast<typename GradientSumT::ValueT*>(d_node_hist),
        page->matrix.info.n_bins * (sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT)));
    reducer->Synchronize();

    monitor.StopCuda("AllReduce");
  }

  /**
   * \brief Build GPU local histograms for the left and right child of some parent node
   */
  void BuildHistLeftRight(const ExpandEntry &candidate, int nidx_left, int nidx_right) {
    auto build_hist_nidx = nidx_left;
    auto subtraction_trick_nidx = nidx_right;

    // Decide whether to build the left histogram or right histogram
    // Use sum of Hessian as a heuristic to select node with fewest training instances
    bool fewer_right = candidate.split.right_sum.GetHess() < candidate.split.left_sum.GetHess();
    if (fewer_right) {
      std::swap(build_hist_nidx, subtraction_trick_nidx);
    }

    this->BuildHist(build_hist_nidx);

    // Check whether we can use the subtraction trick to calculate the other
    bool do_subtraction_trick = this->CanDoSubtractionTrick(
         candidate.nid, build_hist_nidx, subtraction_trick_nidx);

    if (!do_subtraction_trick) {
      // Calculate other histogram manually
      this->BuildHist(subtraction_trick_nidx);
    }
  }

  /**
   * \brief AllReduce GPU histograms for the left and right child of some parent node.
   */
  void ReduceHistLeftRight(const ExpandEntry& candidate,
                           int nidx_left,
                           int nidx_right,
                           dh::AllReducer* reducer) {
    auto build_hist_nidx = nidx_left;
    auto subtraction_trick_nidx = nidx_right;

    // Decide whether to build the left histogram or right histogram
    // Use sum of Hessian as a heuristic to select node with fewest training instances
    bool fewer_right = candidate.split.right_sum.GetHess() < candidate.split.left_sum.GetHess();
    if (fewer_right) {
      std::swap(build_hist_nidx, subtraction_trick_nidx);
    }

    this->AllReduceHist(build_hist_nidx, reducer);

    // Check whether we can use the subtraction trick to calculate the other
    bool do_subtraction_trick = this->CanDoSubtractionTrick(
        candidate.nid, build_hist_nidx, subtraction_trick_nidx);

    if (do_subtraction_trick) {
      // Calculate other histogram using subtraction trick
      this->SubtractionTrick(candidate.nid, build_hist_nidx,
                             subtraction_trick_nidx);
    } else {
      // Calculate other histogram manually
      this->AllReduceHist(subtraction_trick_nidx, reducer);
    }
  }

  void ApplySplit(const ExpandEntry& candidate, RegTree* p_tree) {
    RegTree& tree = *p_tree;

    GradStats left_stats{};
    left_stats.Add(candidate.split.left_sum);
    GradStats right_stats{};
    right_stats.Add(candidate.split.right_sum);
    GradStats parent_sum{};
    parent_sum.Add(left_stats);
    parent_sum.Add(right_stats);
    node_value_constraints.resize(tree.GetNodes().size());
    auto base_weight = node_value_constraints[candidate.nid].CalcWeight(param, parent_sum);
    auto left_weight =
        node_value_constraints[candidate.nid].CalcWeight(param, left_stats)*param.learning_rate;
    auto right_weight =
        node_value_constraints[candidate.nid].CalcWeight(param, right_stats)*param.learning_rate;
    tree.ExpandNode(candidate.nid, candidate.split.findex,
                    candidate.split.fvalue, candidate.split.dir == kLeftDir,
                    base_weight, left_weight, right_weight,
                    candidate.split.loss_chg, parent_sum.sum_hess);
    // Set up child constraints
    node_value_constraints.resize(tree.GetNodes().size());
    node_value_constraints[candidate.nid].SetChild(
        param, tree[candidate.nid].SplitIndex(), left_stats, right_stats,
        &node_value_constraints[tree[candidate.nid].LeftChild()],
        &node_value_constraints[tree[candidate.nid].RightChild()]);
    node_sum_gradients[tree[candidate.nid].LeftChild()] =
        candidate.split.left_sum;
    node_sum_gradients[tree[candidate.nid].RightChild()] =
        candidate.split.right_sum;

    interaction_constraints.Split(candidate.nid, tree[candidate.nid].SplitIndex(),
                                  tree[candidate.nid].LeftChild(),
                                  tree[candidate.nid].RightChild());
  }

  void InitRoot(RegTree* p_tree, HostDeviceVector<GradientPair>* gpair_all, DMatrix* p_fmat,
                dh::AllReducer* reducer, int64_t num_columns) {
    constexpr int kRootNIdx = 0;

    const auto &gpair = gpair_all->DeviceSpan();

    dh::SumReduction(temp_memory, gpair, node_sum_gradients_d,
                     gpair.size());
    reducer->AllReduceSum(
        reinterpret_cast<float*>(node_sum_gradients_d.data()),
        reinterpret_cast<float*>(node_sum_gradients_d.data()), 2);
    reducer->Synchronize();
    dh::safe_cuda(hipMemcpy(node_sum_gradients.data(),
                             node_sum_gradients_d.data(), sizeof(GradientPair),
                             hipMemcpyDeviceToHost));

    this->BuildHistBatches(kRootNIdx, p_fmat);
    this->AllReduceHist(kRootNIdx, reducer);

    // Remember root stats
    p_tree->Stat(kRootNIdx).sum_hess = node_sum_gradients[kRootNIdx].GetHess();
    auto weight = CalcWeight(param, node_sum_gradients[kRootNIdx]);
    p_tree->Stat(kRootNIdx).base_weight = weight;
    (*p_tree)[kRootNIdx].SetLeaf(param.learning_rate * weight);

    // Initialise root constraint
    node_value_constraints.resize(p_tree->GetNodes().size());

    // Generate first split
    auto split = this->EvaluateSplits({kRootNIdx}, *p_tree, num_columns);
    qexpand->push(
        ExpandEntry(kRootNIdx, p_tree->GetDepth(kRootNIdx), split.at(0), 0));
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair_all, DMatrix* p_fmat,
                  RegTree* p_tree, dh::AllReducer* reducer) {
    auto& tree = *p_tree;

    monitor.StartCuda("Reset");
    this->Reset(gpair_all, p_fmat->Info().num_col_);
    monitor.StopCuda("Reset");

    monitor.StartCuda("InitRoot");
    this->InitRoot(p_tree, gpair_all, p_fmat, reducer, p_fmat->Info().num_col_);
    monitor.StopCuda("InitRoot");

    auto timestamp = qexpand->size();
    auto num_leaves = 1;

    while (!qexpand->empty()) {
      ExpandEntry candidate = qexpand->top();
      qexpand->pop();
      if (!candidate.IsValid(param, num_leaves)) {
        continue;
      }
      this->ApplySplit(candidate, p_tree);

      num_leaves++;

      int left_child_nidx = tree[candidate.nid].LeftChild();
      int right_child_nidx = tree[candidate.nid].RightChild();
      // Only create child entries if needed
      if (ExpandEntry::ChildIsValid(param, tree.GetDepth(left_child_nidx), num_leaves)) {
        for (auto& batch : p_fmat->GetBatches<EllpackPage>(batch_param)) {
          page = batch.Impl();

          monitor.StartCuda("UpdatePosition");
          this->UpdatePosition(candidate.nid, (*p_tree)[candidate.nid]);
          monitor.StopCuda("UpdatePosition");

          monitor.StartCuda("BuildHist");
          this->BuildHistLeftRight(candidate, left_child_nidx, right_child_nidx);
          monitor.StopCuda("BuildHist");
        }
        monitor.StartCuda("ReduceHist");
        this->ReduceHistLeftRight(candidate, left_child_nidx, right_child_nidx, reducer);
        monitor.StopCuda("ReduceHist");

        monitor.StartCuda("EvaluateSplits");
        auto splits = this->EvaluateSplits({left_child_nidx, right_child_nidx},
                                           *p_tree, p_fmat->Info().num_col_);
        monitor.StopCuda("EvaluateSplits");

        qexpand->push(ExpandEntry(left_child_nidx,
                                   tree.GetDepth(left_child_nidx), splits.at(0),
                                   timestamp++));
        qexpand->push(ExpandEntry(right_child_nidx,
                                   tree.GetDepth(right_child_nidx),
                                   splits.at(1), timestamp++));
      }
    }

    monitor.StartCuda("FinalisePosition");
    this->FinalisePosition(p_tree, p_fmat);
    monitor.StopCuda("FinalisePosition");
  }
};

template <typename GradientSumT>
inline void GPUHistMakerDevice<GradientSumT>::InitHistogram() {
  CHECK(!(param.max_leaves == 0 && param.max_depth == 0))
      << "Max leaves and max depth cannot both be unconstrained for "
      "gpu_hist.";

  int max_nodes =
      param.max_leaves > 0 ? param.max_leaves * 2 : MaxNodesDepth(param.max_depth);

  ba.Allocate(device_id,
              &gpair, n_rows,
              &prediction_cache, n_rows,
              &node_sum_gradients_d, max_nodes,
              &monotone_constraints, param.monotone_constraints.size());

  dh::CopyVectorToDeviceSpan(monotone_constraints, param.monotone_constraints);

  node_sum_gradients.resize(max_nodes);

  // check if we can use shared memory for building histograms
  // (assuming atleast we need 2 CTAs per SM to maintain decent latency
  // hiding)
  auto histogram_size = sizeof(GradientSumT) * page->matrix.info.n_bins;
  auto max_smem = dh::MaxSharedMemory(device_id);
  if (histogram_size <= max_smem) {
    use_shared_memory_histograms = true;
  }

  // Init histogram
  hist.Init(device_id, page->matrix.info.n_bins);
}

template <typename GradientSumT>
class GPUHistMakerSpecialised {
 public:
  GPUHistMakerSpecialised() : initialised_{false}, p_last_fmat_{nullptr} {}
  void Configure(const Args& args, GenericParameter const* generic_param) {
    param_.UpdateAllowUnknown(args);
    generic_param_ = generic_param;
    hist_maker_param_.UpdateAllowUnknown(args);
    dh::CheckComputeCapability();

    monitor_.Init("updater_gpu_hist");
  }

  ~GPUHistMakerSpecialised() {  // NOLINT
    dh::GlobalMemoryLogger().Log();
  }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) {
    monitor_.StartCuda("Update");

    // rescale learning rate according to size of trees
    float lr = param_.learning_rate;
    param_.learning_rate = lr / trees.size();
    ValueConstraint::Init(&param_, dmat->Info().num_col_);
    // build tree
    try {
      for (xgboost::RegTree* tree : trees) {
        this->UpdateTree(gpair, dmat, tree);

        if (hist_maker_param_.debug_synchronize) {
          this->CheckTreesSynchronized(tree);
        }
      }
      dh::safe_cuda(hipGetLastError());
    } catch (const std::exception& e) {
      LOG(FATAL) << "Exception in gpu_hist: " << e.what() << std::endl;
    }

    param_.learning_rate = lr;
    monitor_.StopCuda("Update");
  }

  void InitDataOnce(DMatrix* dmat) {
    device_ = generic_param_->gpu_id;
    CHECK_GE(device_, 0) << "Must have at least one device";
    info_ = &dmat->Info();
    reducer_.Init({device_});

    // Synchronise the column sampling seed
    uint32_t column_sampling_seed = common::GlobalRandom()();
    rabit::Broadcast(&column_sampling_seed, sizeof(column_sampling_seed), 0);

    BatchParam batch_param{
      device_,
      param_.max_bin,
      hist_maker_param_.gpu_batch_nrows,
      generic_param_->gpu_page_size
    };
    auto page = (*dmat->GetBatches<EllpackPage>(batch_param).begin()).Impl();
    dh::safe_cuda(hipSetDevice(device_));
    maker.reset(new GPUHistMakerDevice<GradientSumT>(device_,
                                                     page,
                                                     info_->num_row_,
                                                     param_,
                                                     column_sampling_seed,
                                                     info_->num_col_,
                                                     batch_param));

    monitor_.StartCuda("InitHistogram");
    dh::safe_cuda(hipSetDevice(device_));
    maker->InitHistogram();
    monitor_.StopCuda("InitHistogram");

    p_last_fmat_ = dmat;
    initialised_ = true;
  }

  void InitData(DMatrix* dmat) {
    if (!initialised_) {
      monitor_.StartCuda("InitDataOnce");
      this->InitDataOnce(dmat);
      monitor_.StopCuda("InitDataOnce");
    }
  }

  // Only call this method for testing
  void CheckTreesSynchronized(RegTree* local_tree) const {
    std::string s_model;
    common::MemoryBufferStream fs(&s_model);
    int rank = rabit::GetRank();
    if (rank == 0) {
      local_tree->Save(&fs);
    }
    fs.Seek(0);
    rabit::Broadcast(&s_model, 0);
    RegTree reference_tree {};  // rank 0 tree
    reference_tree.Load(&fs);
    CHECK(*local_tree == reference_tree);
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* p_fmat,
                  RegTree* p_tree) {
    monitor_.StartCuda("InitData");
    this->InitData(p_fmat);
    monitor_.StopCuda("InitData");

    gpair->SetDevice(device_);
    maker->UpdateTree(gpair, p_fmat, p_tree, &reducer_);
  }

  bool UpdatePredictionCache(const DMatrix* data, HostDeviceVector<bst_float>* p_out_preds) {
    if (maker == nullptr || p_last_fmat_ == nullptr || p_last_fmat_ != data) {
      return false;
    }
    monitor_.StartCuda("UpdatePredictionCache");
    p_out_preds->SetDevice(device_);
    maker->UpdatePredictionCache(p_out_preds->DevicePointer());
    monitor_.StopCuda("UpdatePredictionCache");
    return true;
  }

  TrainParam param_;   // NOLINT
  MetaInfo* info_{};   // NOLINT

  std::unique_ptr<GPUHistMakerDevice<GradientSumT>> maker;  // NOLINT

 private:
  bool initialised_;

  GPUHistMakerTrainParam hist_maker_param_;
  GenericParameter const* generic_param_;

  dh::AllReducer reducer_;

  DMatrix* p_last_fmat_;
  int device_{-1};

  common::Monitor monitor_;
};

class GPUHistMaker : public TreeUpdater {
 public:
  void Configure(const Args& args) override {
    // Used in test to count how many configurations are performed
    LOG(DEBUG) << "[GPU Hist]: Configure";
    hist_maker_param_.UpdateAllowUnknown(args);
    // The passed in args can be empty, if we simply purge the old maker without
    // preserving parameters then we can't do Update on it.
    TrainParam param;
    if (float_maker_) {
      param = float_maker_->param_;
    } else if (double_maker_) {
      param = double_maker_->param_;
    }
    if (hist_maker_param_.single_precision_histogram) {
      float_maker_.reset(new GPUHistMakerSpecialised<GradientPair>());
      float_maker_->param_ = param;
      float_maker_->Configure(args, tparam_);
    } else {
      double_maker_.reset(new GPUHistMakerSpecialised<GradientPairPrecise>());
      double_maker_->param_ = param;
      double_maker_->Configure(args, tparam_);
    }
  }

  void LoadConfig(Json const& in) override {
    auto const& config = get<Object const>(in);
    fromJson(config.at("gpu_hist_train_param"), &this->hist_maker_param_);
    if (hist_maker_param_.single_precision_histogram) {
      float_maker_.reset(new GPUHistMakerSpecialised<GradientPair>());
      fromJson(config.at("train_param"), &float_maker_->param_);
    } else {
      double_maker_.reset(new GPUHistMakerSpecialised<GradientPairPrecise>());
      fromJson(config.at("train_param"), &double_maker_->param_);
    }
  }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["gpu_hist_train_param"] = toJson(hist_maker_param_);
    if (hist_maker_param_.single_precision_histogram) {
      out["train_param"] = toJson(float_maker_->param_);
    } else {
      out["train_param"] = toJson(double_maker_->param_);
    }
  }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) override {
    if (hist_maker_param_.single_precision_histogram) {
      float_maker_->Update(gpair, dmat, trees);
    } else {
      double_maker_->Update(gpair, dmat, trees);
    }
  }

  bool UpdatePredictionCache(
      const DMatrix* data, HostDeviceVector<bst_float>* p_out_preds) override {
    if (hist_maker_param_.single_precision_histogram) {
      return float_maker_->UpdatePredictionCache(data, p_out_preds);
    } else {
      return double_maker_->UpdatePredictionCache(data, p_out_preds);
    }
  }

  char const* Name() const override {
    return "grow_gpu_hist";
  }

 private:
  GPUHistMakerTrainParam hist_maker_param_;
  std::unique_ptr<GPUHistMakerSpecialised<GradientPair>> float_maker_;
  std::unique_ptr<GPUHistMakerSpecialised<GradientPairPrecise>> double_maker_;
};

#if !defined(GTEST_TEST)
XGBOOST_REGISTER_TREE_UPDATER(GPUHistMaker, "grow_gpu_hist")
    .describe("Grow tree with GPU.")
    .set_body([]() { return new GPUHistMaker(); });
#endif  // !defined(GTEST_TEST)

}  // namespace tree
}  // namespace xgboost
