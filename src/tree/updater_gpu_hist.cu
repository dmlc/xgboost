#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2018 XGBoost contributors
 */
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <xgboost/tree_updater.h>
#include <algorithm>
#include <cmath>
#include <memory>
#include <limits>
#include <queue>
#include <utility>
#include <vector>
#include "../common/common.h"
#include "../common/compressed_iterator.h"
#include "../common/device_helpers.cuh"
#include "../common/hist_util.h"
#include "../common/host_device_vector.h"
#include "../common/timer.h"
#include "../common/span.h"
#include "param.h"
#include "updater_gpu_common.cuh"

namespace xgboost {
namespace tree {

DMLC_REGISTRY_FILE_TAG(updater_gpu_hist);

using GradientPairSumT = GradientPairPrecise;

/*!
 * \brief
 *
 * \tparam ReduceT     BlockReduce Type.
 * \tparam TempStorage Cub Shared memory
 *
 * \param begin
 * \param end
 * \param temp_storage Shared memory for intermediate result.
 */
template <int BLOCK_THREADS, typename ReduceT, typename TempStorageT>
__device__ GradientPairSumT ReduceFeature(const GradientPairSumT* begin,
                                          const GradientPairSumT* end,
                                          TempStorageT* temp_storage) {
  __shared__ hipcub::Uninitialized<GradientPairSumT> uninitialized_sum;
  GradientPairSumT& shared_sum = uninitialized_sum.Alias();

  GradientPairSumT local_sum = GradientPairSumT();
  // For loop sums features into one block size
  for (auto itr = begin; itr < end; itr += BLOCK_THREADS) {
    bool thread_active = itr + threadIdx.x < end;
    // Scan histogram
    GradientPairSumT bin = thread_active ? *(itr + threadIdx.x) : GradientPairSumT();
    local_sum += bin;
  }
  local_sum = ReduceT(temp_storage->sum_reduce).Reduce(local_sum, hipcub::Sum());
  // Reduction result is stored in thread 0.
  if (threadIdx.x == 0) {
    shared_sum = local_sum;
  }
  __syncthreads();
  return shared_sum;
}

/*! \brief Find the thread with best gain. */
template <int BLOCK_THREADS, typename ReduceT, typename scan_t,
          typename max_ReduceT, typename TempStorageT>
__device__ void EvaluateFeature(
    int fidx,
    const GradientPairSumT* hist,

    const uint32_t* feature_segments,  // cut.row_ptr
    float min_fvalue,                  // cut.min_value
    const float* gidx_fvalue_map,      // cut.cut

    DeviceSplitCandidate* best_split,  // shared memory storing best split
    const DeviceNodeStats& node,
    const GPUTrainingParam& param,
    TempStorageT* temp_storage,  // temp memory for cub operations
    int constraint,              // monotonic_constraints
    const ValueConstraint& value_constraint) {
  // Use pointer from cut to indicate begin and end of bins for each feature.
  uint32_t gidx_begin = feature_segments[fidx];    // begining bin
  uint32_t gidx_end = feature_segments[fidx + 1];  // end bin for i^th feature

  // Sum histogram bins for current feature
  GradientPairSumT const feature_sum = ReduceFeature<BLOCK_THREADS, ReduceT>(
      hist + gidx_begin, hist + gidx_end, temp_storage);

  GradientPairSumT const parent_sum = GradientPairSumT(node.sum_gradients);
  GradientPairSumT const missing = parent_sum - feature_sum;
  float const null_gain = -std::numeric_limits<bst_float>::infinity();

  SumCallbackOp<GradientPairSumT> prefix_op =
      SumCallbackOp<GradientPairSumT>();
  for (int scan_begin = gidx_begin; scan_begin < gidx_end;
       scan_begin += BLOCK_THREADS) {
    bool thread_active = (scan_begin + threadIdx.x) < gidx_end;

    // Gradient value for current bin.
    GradientPairSumT bin =
        thread_active ? hist[scan_begin + threadIdx.x] : GradientPairSumT();
    scan_t(temp_storage->scan).ExclusiveScan(bin, bin, hipcub::Sum(), prefix_op);

    // Whether the gradient of missing values is put to the left side.
    bool missing_left = true;
    float gain = null_gain;
    if (thread_active) {
      gain = LossChangeMissing(bin, missing, parent_sum, node.root_gain, param,
                               constraint, value_constraint, missing_left);
    }

    __syncthreads();

    // Find thread with best gain
    hipcub::KeyValuePair<int, float> tuple(threadIdx.x, gain);
    hipcub::KeyValuePair<int, float> best =
        max_ReduceT(temp_storage->max_reduce).Reduce(tuple, hipcub::ArgMax());

    __shared__ hipcub::KeyValuePair<int, float> block_max;
    if (threadIdx.x == 0) {
      block_max = best;
    }

    __syncthreads();

    // Best thread updates split
    if (threadIdx.x == block_max.key) {
      int gidx = scan_begin + threadIdx.x;
      float fvalue =
          gidx == gidx_begin ? min_fvalue : gidx_fvalue_map[gidx - 1];
      GradientPairSumT left = missing_left ? bin + missing : bin;
      GradientPairSumT right = parent_sum - left;
      best_split->Update(gain, missing_left ? kLeftDir : kRightDir,
                         fvalue, fidx,
                         GradientPair(left),
                         GradientPair(right),
                         param);
    }
    __syncthreads();
  }
}

template <int BLOCK_THREADS>
__global__ void EvaluateSplitKernel(
    const GradientPairSumT* d_hist,  // histogram for gradients
    uint64_t n_features,
    int* feature_set,  // Selected features
    DeviceNodeStats node,

    const uint32_t* d_feature_segments,  // row_ptr form HistCutMatrix
    const float* d_fidx_min_map,         // min_value
    const float* d_gidx_fvalue_map,      // cut

    GPUTrainingParam gpu_param,
    DeviceSplitCandidate* d_split,  // resulting split
    ValueConstraint value_constraint,
    int* d_monotonic_constraints) {
  // KeyValuePair here used as threadIdx.x -> gain_value
  typedef hipcub::KeyValuePair<int, float> ArgMaxT;
  typedef hipcub::BlockScan<
    GradientPairSumT, BLOCK_THREADS, hipcub::BLOCK_SCAN_WARP_SCANS> BlockScanT;
  typedef hipcub::BlockReduce<ArgMaxT, BLOCK_THREADS> MaxReduceT;

  typedef hipcub::BlockReduce<GradientPairSumT, BLOCK_THREADS> SumReduceT;

  union TempStorage {
    typename BlockScanT::TempStorage scan;
    typename MaxReduceT::TempStorage max_reduce;
    typename SumReduceT::TempStorage sum_reduce;
  };

  // Aligned && shared storage for best_split
  __shared__ hipcub::Uninitialized<DeviceSplitCandidate> uninitialized_split;
  DeviceSplitCandidate& best_split = uninitialized_split.Alias();
  __shared__ TempStorage temp_storage;

  if (threadIdx.x == 0) {
    best_split = DeviceSplitCandidate();
  }

  __syncthreads();

  // One block for each feature. Features are sampled, so fidx != blockIdx.x
  int fidx = feature_set[blockIdx.x];
  int constraint = d_monotonic_constraints[fidx];
  EvaluateFeature<BLOCK_THREADS, SumReduceT, BlockScanT, MaxReduceT>(
      fidx,
      d_hist,

      d_feature_segments,
      d_fidx_min_map[fidx],
      d_gidx_fvalue_map,

      &best_split,
      node,
      gpu_param,
      &temp_storage,
      constraint,
      value_constraint);

  __syncthreads();

  if (threadIdx.x == 0) {
    // Record best loss for each feature
    d_split[fidx] = best_split;
  }
}

// Find a gidx value for a given feature otherwise return -1 if not found
template <typename GidxIterT>
__device__ int BinarySearchRow(bst_uint begin, bst_uint end, GidxIterT data,
                               int const fidx_begin, int const fidx_end) {
  bst_uint previous_middle = UINT32_MAX;
  while (end != begin) {
    auto middle = begin + (end - begin) / 2;
    if (middle == previous_middle) {
      break;
    }
    previous_middle = middle;

    auto gidx = data[middle];

    if (gidx >= fidx_begin && gidx < fidx_end) {
      return gidx;
    } else if (gidx < fidx_begin) {
      begin = middle;
    } else {
      end = middle;
    }
  }
  // Value is missing
  return -1;
}

/**
 * \struct  DeviceHistogram
 *
 * \summary Data storage for node histograms on device. Automatically expands.
 *
 * \author  Rory
 * \date    28/07/2018
 */
struct DeviceHistogram {
  /*! \brief Map nidx to starting index of its histogram. */
  std::map<int, size_t> nidx_map;
  thrust::device_vector<GradientPairSumT::ValueT> data;
  const size_t kStopGrowingSize = 1 << 26;  // Do not grow beyond this size
  int n_bins;
  int device_idx;

  void Init(int device_idx, int n_bins) {
    this->n_bins = n_bins;
    this->device_idx = device_idx;
  }

  void Reset() {
    dh::safe_cuda(hipSetDevice(device_idx));
    data.resize(0);
    nidx_map.clear();
  }

  bool HistogramExists(int nidx) {
    return nidx_map.find(nidx) != nidx_map.end();
  }

  void AllocateHistogram(int nidx) {
    if (HistogramExists(nidx)) return;

    if (data.size() > kStopGrowingSize) {
      // Recycle histogram memory
      std::pair<int, size_t> old_entry = *nidx_map.begin();
      nidx_map.erase(old_entry.first);
      dh::safe_cuda(hipMemset(data.data().get() + old_entry.second, 0,
                               n_bins * sizeof(GradientPairSumT)));
      nidx_map[nidx] = old_entry.second;
    } else {
      // Append new node histogram
      nidx_map[nidx] = data.size();
      dh::safe_cuda(hipSetDevice(device_idx));
      // x 2: Hess and Grad.
      data.resize(data.size() + (n_bins * 2));
    }
  }

  /**
   * \summary   Return pointer to histogram memory for a given node.
   * \param nidx    Tree node index.
   * \return    hist pointer.
   */
  GradientPairSumT* GetHistPtr(int nidx) {
    CHECK(this->HistogramExists(nidx));
    auto ptr = data.data().get() + nidx_map[nidx];
    return reinterpret_cast<GradientPairSumT*>(ptr);
  }
};

struct CalcWeightTrainParam {
  float min_child_weight;
  float reg_alpha;
  float reg_lambda;
  float max_delta_step;
  float learning_rate;
  XGBOOST_DEVICE explicit CalcWeightTrainParam(const TrainParam& p)
      : min_child_weight(p.min_child_weight),
        reg_alpha(p.reg_alpha),
        reg_lambda(p.reg_lambda),
        max_delta_step(p.max_delta_step),
        learning_rate(p.learning_rate) {}
};

// Bin each input data entry, store the bin indices in compressed form.
__global__ void compress_bin_ellpack_k(
    common::CompressedBufferWriter wr,
    common::CompressedByteT* __restrict__ buffer,  // gidx_buffer
    const size_t* __restrict__ row_ptrs,           // row offset of input data
    const Entry* __restrict__ entries,      // One batch of input data
    const float* __restrict__ cuts,         // HistCutMatrix::cut
    const uint32_t* __restrict__ cut_rows,  // HistCutMatrix::row_ptrs
    size_t base_row,                        // batch_row_begin
    size_t n_rows,
    // row_ptr_begin: row_offset[base_row], the start position of base_row
    size_t row_ptr_begin,
    size_t row_stride,
    unsigned int null_gidx_value) {
  size_t irow = threadIdx.x + blockIdx.x * blockDim.x;
  int ifeature = threadIdx.y + blockIdx.y * blockDim.y;
  if (irow >= n_rows || ifeature >= row_stride)
    return;
  int row_length = static_cast<int>(row_ptrs[irow + 1] - row_ptrs[irow]);
  unsigned int bin = null_gidx_value;
  if (ifeature < row_length) {
    Entry entry = entries[row_ptrs[irow] - row_ptr_begin + ifeature];
    int feature = entry.index;
    float fvalue = entry.fvalue;
    // {feature_cuts, ncuts} forms the array of cuts of `feature'.
    const float *feature_cuts = &cuts[cut_rows[feature]];
    int ncuts = cut_rows[feature + 1] - cut_rows[feature];
    // Assigning the bin in current entry.
    // S.t.: fvalue < feature_cuts[bin]
    bin = dh::UpperBound(feature_cuts, ncuts, fvalue);
    if (bin >= ncuts)
      bin = ncuts - 1;
    // Add the number of bins in previous features.
    bin += cut_rows[feature];
  }
  // Write to gidx buffer.
  wr.AtomicWriteSymbol(buffer, bin, (irow + base_row) * row_stride + ifeature);
}

__global__ void sharedMemHistKernel(size_t row_stride,
                                    const bst_uint* d_ridx,
                                    common::CompressedIterator<uint32_t> d_gidx,
                                    int null_gidx_value,
                                    GradientPairSumT* d_node_hist,
                                    const GradientPair* d_gpair,
                                    size_t segment_begin,
                                    size_t n_elements) {
  extern __shared__ char smem[];
  GradientPairSumT* smem_arr = reinterpret_cast<GradientPairSumT*>(smem); // NOLINT
  for (auto i : dh::BlockStrideRange(0, null_gidx_value)) {
    smem_arr[i] = GradientPairSumT();
  }
  __syncthreads();
  for (auto idx : dh::GridStrideRange(static_cast<size_t>(0), n_elements)) {
    int ridx = d_ridx[idx / row_stride + segment_begin];
    int gidx = d_gidx[ridx * row_stride + idx % row_stride];
    if (gidx != null_gidx_value) {
      AtomicAddGpair(smem_arr + gidx, d_gpair[ridx]);
    }
  }
  __syncthreads();
  for (auto i : dh::BlockStrideRange(0, null_gidx_value)) {
    AtomicAddGpair(d_node_hist + i, smem_arr[i]);
  }
}

struct Segment {
  size_t begin;
  size_t end;

  Segment() : begin(0), end(0) {}

  Segment(size_t begin, size_t end) : begin(begin), end(end) {
    CHECK_GE(end, begin);
  }
  size_t Size() const { return end - begin; }
};

struct DeviceShard;

struct GPUHistBuilderBase {
 public:
  virtual void Build(DeviceShard* shard, int idx) = 0;
};

// Manage memory for a single GPU
struct DeviceShard {
  int device_idx;
  /*! \brief Device index counting from param.gpu_id */
  int normalised_device_idx;
  dh::BulkAllocator<dh::MemoryType::kDevice> ba;

  /*! \brief HistCutMatrix stored in device. */
  struct DeviceHistCutMatrix {
    /*! \brief row_ptr form HistCutMatrix. */
    dh::DVec<uint32_t> feature_segments;
    /*! \brief minimum value for each feature. */
    dh::DVec<bst_float> min_fvalue;
    /*! \brief Cut. */
    dh::DVec<bst_float> gidx_fvalue_map;
  } cut_;

  /*! \brief Range of rows for each node. */
  std::vector<Segment> ridx_segments;
  DeviceHistogram hist;

  /*! \brief global index of histogram, which is stored in ELLPack format. */
  dh::DVec<common::CompressedByteT> gidx_buffer;
  /*! \brief row length for ELLPack. */
  size_t row_stride;
  common::CompressedIterator<uint32_t> gidx;

  /*! \brief  Row indices relative to this shard, necessary for sorting rows. */
  dh::DVec2<bst_uint> ridx;
  /*! \brief Gradient pair for each row. */
  dh::DVec<GradientPair> gpair;

  /*! \brief The last histogram index. */
  int null_gidx_value;

  dh::DVec2<int> position;

  dh::DVec<int> monotone_constraints;
  dh::DVec<bst_float> prediction_cache;

  /*! \brief Sum gradient for each node. */
  std::vector<GradientPair> node_sum_gradients;
  dh::DVec<GradientPair> node_sum_gradients_d;
  /*! \brief row offset in SparsePage (the input data). */
  thrust::device_vector<size_t> row_ptrs;
  /*! The row offset for this shard. */
  bst_uint row_begin_idx;
  bst_uint row_end_idx;
  bst_uint n_rows;
  int n_bins;

  TrainParam param;
  bool prediction_cache_initialised;

  // FIXME: Remove this
  int64_t* tmp_pinned;  // Small amount of staging memory

  // Used to process nodes concurrently
  std::vector<hipStream_t> streams;

  dh::CubMemory temp_memory;

  std::unique_ptr<GPUHistBuilderBase> hist_builder;

  // TODO(canonizer): do add support multi-batch DMatrix here
  DeviceShard(int device_idx, int normalised_device_idx,
              bst_uint row_begin, bst_uint row_end, TrainParam _param) :
    device_idx(device_idx),
    normalised_device_idx(normalised_device_idx),
    row_begin_idx(row_begin),
    row_end_idx(row_end),
    row_stride(0),
    n_rows(row_end - row_begin),
    n_bins(0),
    null_gidx_value(0),
    param(_param),
    prediction_cache_initialised(false),
    tmp_pinned(nullptr) {}

  /* Init row_ptrs and row_stride */
  void InitRowPtrs(const SparsePage& row_batch) {
    dh::safe_cuda(hipSetDevice(device_idx));
    const auto& offset_vec = row_batch.offset.HostVector();
    row_ptrs.resize(n_rows + 1);
    thrust::copy(offset_vec.data() + row_begin_idx,
                 offset_vec.data() + row_end_idx + 1,
                 row_ptrs.begin());
    auto row_iter = row_ptrs.begin();
    // find the maximum row size for converting to ELLPack
    auto get_size = [=] __device__(size_t row) {
      return row_iter[row + 1] - row_iter[row];
    }; // NOLINT

    auto counting = thrust::make_counting_iterator(size_t(0));
    using TransformT = thrust::transform_iterator<decltype(get_size),
      decltype(counting), size_t>;
    TransformT row_size_iter = TransformT(counting, get_size);
    row_stride = thrust::reduce(row_size_iter, row_size_iter + n_rows, 0,
                                thrust::maximum<size_t>());
  }

  /*
     Init:
     n_bins, null_gidx_value, gidx_buffer, row_ptrs, gidx, gidx_fvalue_map,
     min_fvalue, feature_segments, node_sum_gradients, ridx_segments,
     hist
  */
  void InitCompressedData(
      const common::HistCutMatrix& hmat, const SparsePage& row_batch);

  void CreateHistIndices(const SparsePage& row_batch);

  ~DeviceShard() {
    for (auto& stream : streams) {
      dh::safe_cuda(hipStreamDestroy(stream));
    }
    dh::safe_cuda(hipHostFree(tmp_pinned));
  }

  // Get vector of at least n initialised streams
  std::vector<hipStream_t>& GetStreams(int n) {
    if (n > streams.size()) {
      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamDestroy(stream));
      }

      streams.clear();
      streams.resize(n);

      for (auto& stream : streams) {
        dh::safe_cuda(hipStreamCreate(&stream));
      }
    }

    return streams;
  }

  // Reset values for each update iteration
  void Reset(HostDeviceVector<GradientPair>* dh_gpair) {
    dh::safe_cuda(hipSetDevice(device_idx));
    position.CurrentDVec().Fill(0);
    std::fill(node_sum_gradients.begin(), node_sum_gradients.end(),
              GradientPair());

    thrust::sequence(ridx.CurrentDVec().tbegin(), ridx.CurrentDVec().tend());

    std::fill(ridx_segments.begin(), ridx_segments.end(), Segment(0, 0));
    ridx_segments.front() = Segment(0, ridx.Size());
    this->gpair.copy(dh_gpair->tcbegin(device_idx), dh_gpair->tcend(device_idx));
    SubsampleGradientPair(&gpair, param.subsample, row_begin_idx);
    hist.Reset();
  }

  void BuildHist(int nidx) {
    hist.AllocateHistogram(nidx);
    hist_builder->Build(this, nidx);
  }

  void SubtractionTrick(int nidx_parent, int nidx_histogram,
                        int nidx_subtraction) {
    auto d_node_hist_parent = hist.GetHistPtr(nidx_parent);
    auto d_node_hist_histogram = hist.GetHistPtr(nidx_histogram);
    auto d_node_hist_subtraction = hist.GetHistPtr(nidx_subtraction);

    dh::LaunchN(device_idx, hist.n_bins, [=] __device__(size_t idx) {
      d_node_hist_subtraction[idx] =
          d_node_hist_parent[idx] - d_node_hist_histogram[idx];
    });
  }

  bool CanDoSubtractionTrick(int nidx_parent, int nidx_histogram,
                             int nidx_subtraction) {
    // Make sure histograms are already allocated
    hist.AllocateHistogram(nidx_subtraction);
    return hist.HistogramExists(nidx_histogram) &&
           hist.HistogramExists(nidx_parent);
  }

  /*! \brief Count how many rows are assigned to left node. */
  __device__ void CountLeft(int64_t* d_count, int val, int left_nidx) {
    unsigned ballot = __ballot(val == left_nidx);
    if (threadIdx.x % 32 == 0) {
      atomicAdd(reinterpret_cast<unsigned long long*>(d_count),    // NOLINT
                static_cast<unsigned long long>(__popc(ballot)));  // NOLINT
    }
  }

  void UpdatePosition(int nidx, int left_nidx, int right_nidx, int fidx,
                      int64_t split_gidx, bool default_dir_left, bool is_dense,
                      int fidx_begin,  // cut.row_ptr[fidx]
                      int fidx_end) {  // cut.row_ptr[fidx + 1]
    dh::safe_cuda(hipSetDevice(device_idx));
    temp_memory.LazyAllocate(sizeof(int64_t));
    int64_t* d_left_count = temp_memory.Pointer<int64_t>();
    dh::safe_cuda(hipMemset(d_left_count, 0, sizeof(int64_t)));
    Segment segment = ridx_segments[nidx];
    bst_uint* d_ridx = ridx.Current();
    int* d_position = position.Current();
    common::CompressedIterator<uint32_t> d_gidx = gidx;
    size_t row_stride = this->row_stride;
    // Launch 1 thread for each row
    dh::LaunchN<1, 512>(
        device_idx, segment.Size(), [=] __device__(bst_uint idx) {
          idx += segment.begin;
          bst_uint ridx = d_ridx[idx];
          auto row_begin = row_stride * ridx;
          auto row_end = row_begin + row_stride;
          auto gidx = -1;
          if (is_dense) {
            // FIXME: Maybe just search the cuts again.
            gidx = d_gidx[row_begin + fidx];
          } else {
            gidx = BinarySearchRow(row_begin, row_end, d_gidx, fidx_begin,
                                   fidx_end);
          }

          // belong to left node or right node.
          int position;
          if (gidx >= 0) {
            // Feature is found
            position = gidx <= split_gidx ? left_nidx : right_nidx;
          } else {
            // Feature is missing
            position = default_dir_left ? left_nidx : right_nidx;
          }

          CountLeft(d_left_count, position, left_nidx);
          d_position[idx] = position;
        });
    dh::safe_cuda(hipMemcpy(tmp_pinned, d_left_count, sizeof(int64_t),
                             hipMemcpyDeviceToHost));
    auto left_count = *tmp_pinned;
    SortPosition(segment, left_nidx, right_nidx);

    ridx_segments[left_nidx] =
        Segment(segment.begin, segment.begin + left_count);
    ridx_segments[right_nidx] =
        Segment(segment.begin + left_count, segment.end);
  }

  /*! \brief Sort row indices according to position. */
  void SortPosition(const Segment& segment, int left_nidx, int right_nidx) {
    int min_bits = 0;
    int max_bits = static_cast<int>(
        std::ceil(std::log2((std::max)(left_nidx, right_nidx) + 1)));

    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs(
        nullptr, temp_storage_bytes,
        position.Current() + segment.begin, position.other() + segment.begin,
        ridx.Current() + segment.begin, ridx.other() + segment.begin,
        segment.Size(), min_bits, max_bits);

    temp_memory.LazyAllocate(temp_storage_bytes);

    hipcub::DeviceRadixSort::SortPairs(
        temp_memory.d_temp_storage, temp_memory.temp_storage_bytes,
        position.Current() + segment.begin, position.other() + segment.begin,
        ridx.Current() + segment.begin, ridx.other() + segment.begin,
        segment.Size(), min_bits, max_bits);
    // Copy back key
    dh::safe_cuda(hipMemcpy(
        position.Current() + segment.begin, position.other() + segment.begin,
        segment.Size() * sizeof(int), hipMemcpyDeviceToDevice));
    // Copy back value
    dh::safe_cuda(hipMemcpy(
        ridx.Current() + segment.begin, ridx.other() + segment.begin,
        segment.Size() * sizeof(bst_uint), hipMemcpyDeviceToDevice));
  }

  void UpdatePredictionCache(bst_float* out_preds_d) {
    dh::safe_cuda(hipSetDevice(device_idx));
    if (!prediction_cache_initialised) {
      dh::safe_cuda(hipMemcpy(
          prediction_cache.Data(), out_preds_d,
          prediction_cache.Size() * sizeof(bst_float), hipMemcpyDefault));
    }
    prediction_cache_initialised = true;

    CalcWeightTrainParam param_d(param);

    dh::safe_cuda(hipMemcpy(node_sum_gradients_d.Data(),
                             node_sum_gradients.data(),
                             sizeof(GradientPair) * node_sum_gradients.size(),
                             hipMemcpyHostToDevice));
    auto d_position = position.Current();
    auto d_ridx = ridx.Current();
    auto d_node_sum_gradients = node_sum_gradients_d.Data();
    auto d_prediction_cache = prediction_cache.Data();

    dh::LaunchN(
        device_idx, prediction_cache.Size(), [=] __device__(int local_idx) {
          int pos = d_position[local_idx];
          bst_float weight = CalcWeight(param_d, d_node_sum_gradients[pos]);
          d_prediction_cache[d_ridx[local_idx]] +=
              weight * param_d.learning_rate;
        });

    dh::safe_cuda(hipMemcpy(
        out_preds_d, prediction_cache.Data(),
        prediction_cache.Size() * sizeof(bst_float), hipMemcpyDefault));
  }
};

struct SharedMemHistBuilder : public GPUHistBuilderBase {
  void Build(DeviceShard* shard, int nidx) override {
    auto segment = shard->ridx_segments[nidx];
    auto segment_begin = segment.begin;
    auto d_node_hist = shard->hist.GetHistPtr(nidx);
    auto d_gidx = shard->gidx;
    auto d_ridx = shard->ridx.Current();
    auto d_gpair = shard->gpair.Data();

    int null_gidx_value = shard->null_gidx_value;
    auto n_elements = segment.Size() * shard->row_stride;

    const size_t smem_size = sizeof(GradientPairSumT) * shard->null_gidx_value;
    const int items_per_thread = 8;
    const int block_threads = 256;
    const int grid_size =
        static_cast<int>(dh::DivRoundUp(n_elements,
                                        items_per_thread * block_threads));
    if (grid_size <= 0) {
      return;
    }
    dh::safe_cuda(hipSetDevice(shard->device_idx));
    sharedMemHistKernel<<<grid_size, block_threads, smem_size>>>
        (shard->row_stride, d_ridx, d_gidx, null_gidx_value, d_node_hist, d_gpair,
         segment_begin, n_elements);
  }
};

struct GlobalMemHistBuilder : public GPUHistBuilderBase {
  void Build(DeviceShard* shard, int nidx) override {
    Segment segment = shard->ridx_segments[nidx];
    GradientPairSumT* d_node_hist = shard->hist.GetHistPtr(nidx);
    common::CompressedIterator<uint32_t> d_gidx = shard->gidx;
    bst_uint* d_ridx = shard->ridx.Current();
    GradientPair* d_gpair = shard->gpair.Data();

    size_t const n_elements = segment.Size() * shard->row_stride;
    size_t const row_stride = shard->row_stride;
    int const null_gidx_value = shard->null_gidx_value;

    dh::LaunchN(shard->device_idx, n_elements, [=] __device__(size_t idx) {
        int ridx = d_ridx[(idx / row_stride) + segment.begin];
        // lookup the index (bin) of histogram.
        int gidx = d_gidx[ridx * row_stride + idx % row_stride];

        if (gidx != null_gidx_value) {
          AtomicAddGpair(d_node_hist + gidx, d_gpair[ridx]);
        }
      });
  }
};

inline void DeviceShard::InitCompressedData(
    const common::HistCutMatrix& hmat, const SparsePage& row_batch) {
  n_bins = hmat.row_ptr.back();
  null_gidx_value = hmat.row_ptr.back();

  int max_nodes =
      param.max_leaves > 0 ? param.max_leaves * 2 : MaxNodesDepth(param.max_depth);

  ba.Allocate(device_idx, param.silent,
              &gpair, n_rows,
              &ridx, n_rows,
              &position, n_rows,
              &prediction_cache, n_rows,
              &node_sum_gradients_d, max_nodes,
              &cut_.feature_segments, hmat.row_ptr.size(),
              &cut_.gidx_fvalue_map, hmat.cut.size(),
              &cut_.min_fvalue, hmat.min_val.size(),
              &monotone_constraints, param.monotone_constraints.size());
  cut_.gidx_fvalue_map = hmat.cut;
  cut_.min_fvalue = hmat.min_val;
  cut_.feature_segments = hmat.row_ptr;
  monotone_constraints = param.monotone_constraints;

  node_sum_gradients.resize(max_nodes);
  ridx_segments.resize(max_nodes);

  dh::safe_cuda(hipSetDevice(device_idx));

  // allocate compressed bin data
  int num_symbols = n_bins + 1;
  // Required buffer size for storing data matrix in ELLPack format.
  size_t compressed_size_bytes =
      common::CompressedBufferWriter::CalculateBufferSize(row_stride * n_rows,
                                                          num_symbols);

  CHECK(!(param.max_leaves == 0 && param.max_depth == 0))
      << "Max leaves and max depth cannot both be unconstrained for "
      "gpu_hist.";
  ba.Allocate(device_idx, param.silent, &gidx_buffer, compressed_size_bytes);
  gidx_buffer.Fill(0);

  int nbits = common::detail::SymbolBits(num_symbols);

  CreateHistIndices(row_batch);

  gidx = common::CompressedIterator<uint32_t>(gidx_buffer.Data(), num_symbols);

  // check if we can use shared memory for building histograms
  // (assuming atleast we need 2 CTAs per SM to maintain decent latency hiding)
  auto histogram_size = sizeof(GradientPairSumT) * null_gidx_value;
  auto max_smem = dh::MaxSharedMemory(device_idx);
  if (histogram_size <= max_smem) {
    hist_builder.reset(new SharedMemHistBuilder);
  } else {
    hist_builder.reset(new GlobalMemHistBuilder);
  }

  // Init histogram
  hist.Init(device_idx, hmat.row_ptr.back());

  dh::safe_cuda(hipHostMalloc(&tmp_pinned, sizeof(int64_t)));
}

inline void DeviceShard::CreateHistIndices(const SparsePage& row_batch) {
  int num_symbols = n_bins + 1;
  // bin and compress entries in batches of rows
  size_t gpu_batch_nrows = std::min
                           (dh::TotalMemory(device_idx) / (16 * row_stride * sizeof(Entry)),
                            static_cast<size_t>(n_rows));
  const std::vector<Entry>& data_vec = row_batch.data.HostVector();

  thrust::device_vector<Entry> entries_d(gpu_batch_nrows * row_stride);
  size_t gpu_nbatches = dh::DivRoundUp(n_rows, gpu_batch_nrows);

  for (size_t gpu_batch = 0; gpu_batch < gpu_nbatches; ++gpu_batch) {
    size_t batch_row_begin = gpu_batch * gpu_batch_nrows;
    size_t batch_row_end = (gpu_batch + 1) * gpu_batch_nrows;
    if (batch_row_end > n_rows) {
      batch_row_end = n_rows;
    }
    size_t batch_nrows = batch_row_end - batch_row_begin;
    // number of entries in this batch.
    size_t n_entries = row_ptrs[batch_row_end] - row_ptrs[batch_row_begin];
    // copy data entries to device.
    dh::safe_cuda
        (hipMemcpy
         (entries_d.data().get(), data_vec.data() + row_ptrs[batch_row_begin],
          n_entries * sizeof(Entry), hipMemcpyDefault));
    const dim3 block3(32, 8, 1);  // 256 threads
    const dim3 grid3(dh::DivRoundUp(n_rows, block3.x),
                     dh::DivRoundUp(row_stride, block3.y), 1);
    compress_bin_ellpack_k<<<grid3, block3>>>
        (common::CompressedBufferWriter(num_symbols),
         gidx_buffer.Data(),
         row_ptrs.data().get() + batch_row_begin,
         entries_d.data().get(),
         cut_.gidx_fvalue_map.Data(), cut_.feature_segments.Data(),
         batch_row_begin, batch_nrows,
         row_ptrs[batch_row_begin],
         row_stride, null_gidx_value);

    dh::safe_cuda(hipGetLastError());
    dh::safe_cuda(hipDeviceSynchronize());
  }

  // free the memory that is no longer needed
  row_ptrs.resize(0);
  row_ptrs.shrink_to_fit();
  entries_d.resize(0);
  entries_d.shrink_to_fit();
}

class GPUHistMaker : public TreeUpdater {
 public:
  struct ExpandEntry;

  GPUHistMaker() : initialised_(false), p_last_fmat_(nullptr) {}
  void Init(
      const std::vector<std::pair<std::string, std::string>>& args) override {
    param_.InitAllowUnknown(args);
    CHECK(param_.n_gpus != 0) << "Must have at least one device";
    n_devices_ = param_.n_gpus;
    dist_ = GPUDistribution::Block(GPUSet::All(param_.n_gpus)
                                   .Normalised(param_.gpu_id));

    dh::CheckComputeCapability();

    if (param_.grow_policy == TrainParam::kLossGuide) {
      qexpand_.reset(new ExpandQueue(LossGuide));
    } else {
      qexpand_.reset(new ExpandQueue(DepthWise));
    }

    monitor_.Init("updater_gpu_hist", param_.debug_verbose);
  }

  void Update(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) override {
    monitor_.Start("Update", dist_.Devices());
    GradStats::CheckInfo(dmat->Info());
    // rescale learning rate according to size of trees
    float lr = param_.learning_rate;
    param_.learning_rate = lr / trees.size();
    ValueConstraint::Init(&param_, dmat->Info().num_col_);
    // build tree
    try {
      for (size_t i = 0; i < trees.size(); ++i) {
        this->UpdateTree(gpair, dmat, trees[i]);
      }
      dh::safe_cuda(hipGetLastError());
    } catch (const std::exception& e) {
      LOG(FATAL) << "Exception in gpu_hist: " << e.what() << std::endl;
    }
    param_.learning_rate = lr;
    monitor_.Stop("Update", dist_.Devices());
  }

  void InitDataOnce(DMatrix* dmat) {
    info_ = &dmat->Info();

    int n_devices = GPUSet::All(param_.n_gpus, info_->num_row_).Size();

    device_list_.resize(n_devices);
    for (int d_idx = 0; d_idx < n_devices; ++d_idx) {
      int device_idx = GPUSet::GetDeviceIdx(param_.gpu_id + d_idx);
      device_list_[d_idx] = device_idx;
    }

    reducer_.Init(device_list_);

    auto batch_iter = dmat->GetRowBatches().begin();
    const SparsePage& batch = *batch_iter;
    // Create device shards
    shards_.resize(n_devices);
    dh::ExecuteIndexShards(&shards_, [&](int i, std::unique_ptr<DeviceShard>& shard) {
        size_t start = dist_.ShardStart(info_->num_row_, i);
        size_t size = dist_.ShardSize(info_->num_row_, i);
        shard = std::unique_ptr<DeviceShard>
          (new DeviceShard(device_list_.at(i), i,
                           start, start + size, param_));
        shard->InitRowPtrs(batch);
      });

    // Find the cuts.
    monitor_.Start("Quantiles", dist_.Devices());
    common::DeviceSketch(batch, *info_, param_, &hmat_);
    n_bins_ = hmat_.row_ptr.back();
    monitor_.Stop("Quantiles", dist_.Devices());

    monitor_.Start("BinningCompression", dist_.Devices());
    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->InitCompressedData(hmat_, batch);
      });
    monitor_.Stop("BinningCompression", dist_.Devices());
    ++batch_iter;
    CHECK(batch_iter.AtEnd()) << "External memory not supported";

    p_last_fmat_ = dmat;
    initialised_ = true;
  }

  void InitData(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat) {
    monitor_.Start("InitDataOnce", dist_.Devices());
    if (!initialised_) {
      this->InitDataOnce(dmat);
    }
    monitor_.Stop("InitDataOnce", dist_.Devices());

    column_sampler_.Init(info_->num_col_, param_.colsample_bylevel, param_.colsample_bytree);

    // Copy gpair & reset memory
    monitor_.Start("InitDataReset", dist_.Devices());

    gpair->Reshard(dist_);
    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->Reset(gpair);
      });
    monitor_.Stop("InitDataReset", dist_.Devices());
  }

  void AllReduceHist(int nidx) {
    reducer_.GroupStart();
    for (auto& shard : shards_) {
      auto d_node_hist = shard->hist.GetHistPtr(nidx);
      reducer_.AllReduceSum(
          shard->normalised_device_idx,
          reinterpret_cast<GradientPairSumT::ValueT*>(d_node_hist),
          reinterpret_cast<GradientPairSumT::ValueT*>(d_node_hist),
          n_bins_ * (sizeof(GradientPairSumT) / sizeof(GradientPairSumT::ValueT)));
    }
    reducer_.GroupEnd();

    reducer_.Synchronize();
  }

  void BuildHistLeftRight(int nidx_parent, int nidx_left, int nidx_right) {
    size_t left_node_max_elements = 0;
    size_t right_node_max_elements = 0;
    for (auto& shard : shards_) {
      left_node_max_elements = (std::max)(
          left_node_max_elements, shard->ridx_segments[nidx_left].Size());
      right_node_max_elements = (std::max)(
          right_node_max_elements, shard->ridx_segments[nidx_right].Size());
    }

    auto build_hist_nidx = nidx_left;
    auto subtraction_trick_nidx = nidx_right;

    if (right_node_max_elements < left_node_max_elements) {
      build_hist_nidx = nidx_right;
      subtraction_trick_nidx = nidx_left;
    }

    // Build histogram for node with the smallest number of training examples
    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->BuildHist(build_hist_nidx);
      });

    this->AllReduceHist(build_hist_nidx);

    // Check whether we can use the subtraction trick to calculate the other
    bool do_subtraction_trick = true;
    for (auto& shard : shards_) {
      do_subtraction_trick &= shard->CanDoSubtractionTrick(
          nidx_parent, build_hist_nidx, subtraction_trick_nidx);
    }

    if (do_subtraction_trick) {
      // Calculate other histogram using subtraction trick
      dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->SubtractionTrick(nidx_parent, build_hist_nidx,
                                subtraction_trick_nidx);
      });
    } else {
      // Calculate other histogram manually
      dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->BuildHist(subtraction_trick_nidx);
      });

      this->AllReduceHist(subtraction_trick_nidx);
    }
  }

  // Returns best loss
  std::vector<DeviceSplitCandidate> EvaluateSplits(
      const std::vector<int>& nidx_set, RegTree* p_tree) {
    size_t const columns = info_->num_col_;
    std::vector<DeviceSplitCandidate> best_splits(nidx_set.size());
    // Every feature is a candidate
    size_t const candidates_size_bytes =
        nidx_set.size() * columns * sizeof(DeviceSplitCandidate);
    // Storage for all candidates from all nodes.
    std::vector<DeviceSplitCandidate> candidate_splits(nidx_set.size() * columns);
    // FIXME: Multi-gpu support?
    // Use first device
    auto& shard = shards_.front();
    dh::safe_cuda(hipSetDevice(shard->device_idx));
    shard->temp_memory.LazyAllocate(candidates_size_bytes);
    auto d_split = shard->temp_memory.Pointer<DeviceSplitCandidate>();

    auto& streams = shard->GetStreams(static_cast<int>(nidx_set.size()));

    // Use streams to process nodes concurrently
    for (auto i = 0; i < nidx_set.size(); i++) {
      auto nidx = nidx_set[i];
      DeviceNodeStats node(shard->node_sum_gradients[nidx], nidx, param_);
      int depth = p_tree->GetDepth(nidx);

      HostDeviceVector<int>& feature_set = column_sampler_.GetFeatureSet(depth);
      feature_set.Reshard(GPUSet::Range(shard->device_idx, 1));
      auto& h_feature_set = feature_set.HostVector();
      // One block for each feature
      int constexpr BLOCK_THREADS = 256;
      EvaluateSplitKernel<BLOCK_THREADS>
          <<<uint32_t(feature_set.Size()), BLOCK_THREADS, 0, streams[i]>>>(
              shard->hist.GetHistPtr(nidx),
              info_->num_col_,
              feature_set.DevicePointer(shard->device_idx),
              node,
              shard->cut_.feature_segments.Data(),
              shard->cut_.min_fvalue.Data(),
              shard->cut_.gidx_fvalue_map.Data(),
              GPUTrainingParam(param_),
              d_split + i * columns,  // split candidate for i^th node.
              node_value_constraints_[nidx],
              shard->monotone_constraints.Data());
    }

    dh::safe_cuda(hipDeviceSynchronize());
    dh::safe_cuda(
        hipMemcpy(candidate_splits.data(), shard->temp_memory.d_temp_storage,
                   candidates_size_bytes, hipMemcpyDeviceToHost));
    for (auto i = 0; i < nidx_set.size(); i++) {
      auto depth = p_tree->GetDepth(nidx_set[i]);
      DeviceSplitCandidate nidx_best;
      for (auto fidx : column_sampler_.GetFeatureSet(depth).HostVector()) {
        DeviceSplitCandidate& candidate =
            candidate_splits[i * columns + fidx];
        nidx_best.Update(candidate, param_);
      }
      best_splits[i] = nidx_best;
    }
    return std::move(best_splits);
  }

  void InitRoot(RegTree* p_tree) {
    constexpr int root_nidx = 0;
    // Sum gradients
    std::vector<GradientPair> tmp_sums(shards_.size());

    dh::ExecuteIndexShards(&shards_, [&](int i, std::unique_ptr<DeviceShard>& shard) {
        dh::safe_cuda(hipSetDevice(shard->device_idx));
      tmp_sums[i] =
        dh::SumReduction(shard->temp_memory, shard->gpair.Data(),
                         shard->gpair.Size());
      });
    GradientPair sum_gradient =
        std::accumulate(tmp_sums.begin(), tmp_sums.end(), GradientPair());

    // Generate root histogram
    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->BuildHist(root_nidx);
      });

    this->AllReduceHist(root_nidx);

    // Remember root stats
    p_tree->Stat(root_nidx).sum_hess = sum_gradient.GetHess();
    auto weight = CalcWeight(param_, sum_gradient);
    p_tree->Stat(root_nidx).base_weight = weight;
    (*p_tree)[root_nidx].SetLeaf(param_.learning_rate * weight);

    // Store sum gradients
    for (auto& shard : shards_) {
      shard->node_sum_gradients[root_nidx] = sum_gradient;
    }

    // Initialise root constraint
    node_value_constraints_.resize(p_tree->GetNodes().size());

    // Generate first split
    auto splits = this->EvaluateSplits({root_nidx}, p_tree);
    qexpand_->push(
        ExpandEntry(root_nidx, p_tree->GetDepth(root_nidx), splits.front(), 0));
  }

  void UpdatePosition(const ExpandEntry& candidate, RegTree* p_tree) {
    int nidx = candidate.nid;
    int left_nidx = (*p_tree)[nidx].LeftChild();
    int right_nidx = (*p_tree)[nidx].RightChild();

    // convert floating-point split_pt into corresponding bin_id
    // split_cond = -1 indicates that split_pt is less than all known cut points
    int64_t split_gidx = -1;
    int64_t fidx = candidate.split.findex;
    bool default_dir_left = candidate.split.dir == kLeftDir;
    uint32_t fidx_begin = hmat_.row_ptr[fidx];
    uint32_t fidx_end = hmat_.row_ptr[fidx + 1];
    // split_gidx = i where i is the i^th bin containing split value.
    for (auto i = fidx_begin; i < fidx_end; ++i) {
      if (candidate.split.fvalue == hmat_.cut[i]) {
        split_gidx = static_cast<int64_t>(i);
      }
    }
    auto is_dense = info_->num_nonzero_ == info_->num_row_ * info_->num_col_;

    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->UpdatePosition(nidx, left_nidx, right_nidx, fidx,
                              split_gidx, default_dir_left,
                              is_dense, fidx_begin, fidx_end);
      });
  }

  void ApplySplit(const ExpandEntry& candidate, RegTree* p_tree) {
    // Add new leaves
    RegTree& tree = *p_tree;
    tree.AddChilds(candidate.nid);
    auto& parent = tree[candidate.nid];
    parent.SetSplit(candidate.split.findex, candidate.split.fvalue,
                    candidate.split.dir == kLeftDir);
    tree.Stat(candidate.nid).loss_chg = candidate.split.loss_chg;

    // Set up child constraints
    node_value_constraints_.resize(tree.GetNodes().size());
    GradStats left_stats(param_);
    left_stats.Add(candidate.split.left_sum);
    GradStats right_stats(param_);
    right_stats.Add(candidate.split.right_sum);
    node_value_constraints_[candidate.nid].SetChild(
        param_, parent.SplitIndex(), left_stats, right_stats,
        &node_value_constraints_[parent.LeftChild()],
        &node_value_constraints_[parent.RightChild()]);

    // Configure left child
    auto left_weight =
        node_value_constraints_[parent.LeftChild()].CalcWeight(param_, left_stats);
    tree[parent.LeftChild()].SetLeaf(left_weight * param_.learning_rate, 0);
    tree.Stat(parent.LeftChild()).base_weight = left_weight;
    tree.Stat(parent.LeftChild()).sum_hess = candidate.split.left_sum.GetHess();

    // Configure right child
    auto right_weight =
        node_value_constraints_[parent.RightChild()].CalcWeight(param_, right_stats);
    tree[parent.RightChild()].SetLeaf(right_weight * param_.learning_rate, 0);
    tree.Stat(parent.RightChild()).base_weight = right_weight;
    tree.Stat(parent.RightChild()).sum_hess = candidate.split.right_sum.GetHess();

    // Store sum gradients
    for (auto& shard : shards_) {
      shard->node_sum_gradients[parent.LeftChild()] = candidate.split.left_sum;
      shard->node_sum_gradients[parent.RightChild()] = candidate.split.right_sum;
    }
    this->UpdatePosition(candidate, p_tree);
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* p_fmat,
                  RegTree* p_tree) {
    auto& tree = *p_tree;

    monitor_.Start("InitData", dist_.Devices());
    this->InitData(gpair, p_fmat);
    monitor_.Stop("InitData", dist_.Devices());
    monitor_.Start("InitRoot", dist_.Devices());
    this->InitRoot(p_tree);
    monitor_.Stop("InitRoot", dist_.Devices());

    auto timestamp = qexpand_->size();
    auto num_leaves = 1;

    while (!qexpand_->empty()) {
      ExpandEntry candidate = qexpand_->top();
      qexpand_->pop();
      if (!candidate.IsValid(param_, num_leaves)) continue;

      monitor_.Start("ApplySplit", dist_.Devices());
      this->ApplySplit(candidate, p_tree);
      monitor_.Stop("ApplySplit", dist_.Devices());
      num_leaves++;

      int left_child_nidx = tree[candidate.nid].LeftChild();
      int right_child_nidx = tree[candidate.nid].RightChild();

      // Only create child entries if needed
      if (ExpandEntry::ChildIsValid(param_, tree.GetDepth(left_child_nidx),
                                    num_leaves)) {
        monitor_.Start("BuildHist", dist_.Devices());
        this->BuildHistLeftRight(candidate.nid, left_child_nidx,
                                 right_child_nidx);
        monitor_.Stop("BuildHist", dist_.Devices());

        monitor_.Start("EvaluateSplits", dist_.Devices());
        auto splits =
            this->EvaluateSplits({left_child_nidx, right_child_nidx}, p_tree);
        qexpand_->push(ExpandEntry(left_child_nidx,
                                   tree.GetDepth(left_child_nidx), splits[0],
                                   timestamp++));
        qexpand_->push(ExpandEntry(right_child_nidx,
                                   tree.GetDepth(right_child_nidx), splits[1],
                                   timestamp++));
        monitor_.Stop("EvaluateSplits", dist_.Devices());
      }
    }
  }

  bool UpdatePredictionCache(
      const DMatrix* data, HostDeviceVector<bst_float>* p_out_preds) override {
    monitor_.Start("UpdatePredictionCache", dist_.Devices());
    if (shards_.empty() || p_last_fmat_ == nullptr || p_last_fmat_ != data)
      return false;
    p_out_preds->Reshard(dist_.Devices());
    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->UpdatePredictionCache(p_out_preds->DevicePointer(shard->device_idx));
      });
    monitor_.Stop("UpdatePredictionCache", dist_.Devices());
    return true;
  }

  struct ExpandEntry {
    int nid;
    int depth;
    DeviceSplitCandidate split;
    uint64_t timestamp;
    ExpandEntry(int nid, int depth, const DeviceSplitCandidate& split,
                uint64_t timestamp)
        : nid(nid), depth(depth), split(split), timestamp(timestamp) {}
    bool IsValid(const TrainParam& param, int num_leaves) const {
      if (split.loss_chg <= kRtEps) return false;
      if (split.left_sum.GetHess() == 0 || split.right_sum.GetHess() == 0)
        return false;
      if (param.max_depth > 0 && depth == param.max_depth) return false;
      if (param.max_leaves > 0 && num_leaves == param.max_leaves) return false;
      return true;
    }

    static bool ChildIsValid(const TrainParam& param, int depth,
                             int num_leaves) {
      if (param.max_depth > 0 && depth >= param.max_depth) return false;
      if (param.max_leaves > 0 && num_leaves >= param.max_leaves) return false;
      return true;
    }

    friend std::ostream& operator<<(std::ostream& os, const ExpandEntry& e) {
      os << "ExpandEntry: \n";
      os << "nidx: " << e.nid << "\n";
      os << "depth: " << e.depth << "\n";
      os << "loss: " << e.split.loss_chg << "\n";
      os << "left_sum: " << e.split.left_sum << "\n";
      os << "right_sum: " << e.split.right_sum << "\n";
      return os;
    }
  };

  inline static bool DepthWise(ExpandEntry lhs, ExpandEntry rhs) {
    if (lhs.depth == rhs.depth) {
      return lhs.timestamp > rhs.timestamp;  // favor small timestamp
    } else {
      return lhs.depth > rhs.depth;  // favor small depth
    }
  }
  inline static bool LossGuide(ExpandEntry lhs, ExpandEntry rhs) {
    if (lhs.split.loss_chg == rhs.split.loss_chg) {
      return lhs.timestamp > rhs.timestamp;  // favor small timestamp
    } else {
      return lhs.split.loss_chg < rhs.split.loss_chg;  // favor large loss_chg
    }
  }
  TrainParam param_;
  common::HistCutMatrix hmat_;
  common::GHistIndexMatrix gmat_;
  MetaInfo* info_;
  bool initialised_;
  int n_devices_;
  int n_bins_;

  std::vector<std::unique_ptr<DeviceShard>> shards_;
  common::ColumnSampler column_sampler_;
  using ExpandQueue = std::priority_queue<ExpandEntry, std::vector<ExpandEntry>,
    std::function<bool(ExpandEntry, ExpandEntry)>>;
  std::unique_ptr<ExpandQueue> qexpand_;
  common::Monitor monitor_;
  dh::AllReducer reducer_;
  std::vector<ValueConstraint> node_value_constraints_;
  std::vector<int> device_list_;

  DMatrix* p_last_fmat_;
  GPUDistribution dist_;
};

XGBOOST_REGISTER_TREE_UPDATER(GPUHistMaker, "grow_gpu_hist")
    .describe("Grow tree with GPU.")
    .set_body([]() { return new GPUHistMaker(); });
}  // namespace tree
}  // namespace xgboost
