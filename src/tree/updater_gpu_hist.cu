#include "hip/hip_runtime.h"
/*!
 * Copyright 2017 XGBoost contributors
 */
#include <xgboost/tree_updater.h>
#include <memory>
#include <utility>
#include <vector>
#include "../common/compressed_iterator.h"
#include "../common/device_helpers.cuh"
#include "../common/hist_util.h"
#include "param.h"
#include "updater_gpu_common.cuh"

namespace xgboost {
namespace tree {

DMLC_REGISTRY_FILE_TAG(updater_gpu_hist);

typedef bst_gpair_integer gpair_sum_t;
static const ncclDataType_t nccl_sum_t = ncclInt64;

// Helper for explicit template specialisation
template <int N>
struct Int {};

struct DeviceGMat {
  dh::dvec<common::compressed_byte_t> gidx_buffer;
  common::CompressedIterator<uint32_t> gidx;
  dh::dvec<size_t> row_ptr;
  void Init(int device_idx, const common::GHistIndexMatrix& gmat,
            bst_ulong element_begin, bst_ulong element_end, bst_ulong row_begin,
            bst_ulong row_end, int n_bins) {
    dh::safe_cuda(hipSetDevice(device_idx));
    CHECK(gidx_buffer.size()) << "gidx_buffer must be externally allocated";
    CHECK_EQ(row_ptr.size(), (row_end - row_begin) + 1)
        << "row_ptr must be externally allocated";

    common::CompressedBufferWriter cbw(n_bins);
    std::vector<common::compressed_byte_t> host_buffer(gidx_buffer.size());
    cbw.Write(host_buffer.data(), gmat.index.begin() + element_begin,
              gmat.index.begin() + element_end);
    gidx_buffer = host_buffer;
    gidx = common::CompressedIterator<uint32_t>(gidx_buffer.data(), n_bins);

    // row_ptr
    dh::safe_cuda(hipMemcpy(row_ptr.data(), gmat.row_ptr.data() + row_begin,
                             row_ptr.size() * sizeof(size_t),
                             hipMemcpyHostToDevice));
    // normalise row_ptr
    size_t start = gmat.row_ptr[row_begin];
    auto d_row_ptr = row_ptr.data();
    dh::launch_n(row_ptr.device_idx(), row_ptr.size(),
                 [=] __device__(size_t idx) { d_row_ptr[idx] -= start; });
  }
};

struct HistHelper {
  gpair_sum_t* d_hist;
  int n_bins;
  __host__ __device__ HistHelper(gpair_sum_t* ptr, int n_bins)
      : d_hist(ptr), n_bins(n_bins) {}

  __device__ void Add(bst_gpair gpair, int gidx, int nidx) const {
    int hist_idx = nidx * n_bins + gidx;

    auto dst_ptr =
        reinterpret_cast<unsigned long long int*>(&d_hist[hist_idx]);  // NOLINT
    gpair_sum_t tmp(gpair.GetGrad(), gpair.GetHess());
    auto src_ptr = reinterpret_cast<gpair_sum_t::value_t*>(&tmp);

    atomicAdd(dst_ptr,
              static_cast<unsigned long long int>(*src_ptr));  // NOLINT
    atomicAdd(dst_ptr + 1,
              static_cast<unsigned long long int>(*(src_ptr + 1)));  // NOLINT
  }
  __device__ gpair_sum_t Get(int gidx, int nidx) const {
    return d_hist[nidx * n_bins + gidx];
  }
};

struct DeviceHist {
  int n_bins;
  dh::dvec<gpair_sum_t> data;

  void Init(int n_bins_in) {
    this->n_bins = n_bins_in;
    CHECK(!data.empty()) << "DeviceHist must be externally allocated";
  }

  void Reset(int device_idx) {
    hipSetDevice(device_idx);
    data.fill(gpair_sum_t());
  }

  HistHelper GetBuilder() { return HistHelper(data.data(), n_bins); }

  gpair_sum_t* GetLevelPtr(int depth) {
    return data.data() + n_nodes(depth - 1) * n_bins;
  }

  int LevelSize(int depth) { return n_bins * n_nodes_level(depth); }
};

template <int BLOCK_THREADS>
__global__ void find_split_kernel(
    const gpair_sum_t* d_level_hist, int* d_feature_segments, int depth,
    int n_features, int n_bins, DeviceNodeStats* d_nodes,
    int nodes_offset_device, float* d_fidx_min_map, float* d_gidx_fvalue_map,
    GPUTrainingParam gpu_param, bool* d_left_child_smallest_temp,
    bool colsample, int* d_feature_flags) {
  typedef hipcub::KeyValuePair<int, float> ArgMaxT;
  typedef hipcub::BlockScan<gpair_sum_t, BLOCK_THREADS, hipcub::BLOCK_SCAN_WARP_SCANS>
      BlockScanT;
  typedef hipcub::BlockReduce<ArgMaxT, BLOCK_THREADS> MaxReduceT;
  typedef hipcub::BlockReduce<gpair_sum_t, BLOCK_THREADS> SumReduceT;

  union TempStorage {
    typename BlockScanT::TempStorage scan;
    typename MaxReduceT::TempStorage max_reduce;
    typename SumReduceT::TempStorage sum_reduce;
  };

  __shared__ hipcub::Uninitialized<DeviceSplitCandidate> uninitialized_split;
  DeviceSplitCandidate& split = uninitialized_split.Alias();
  __shared__ hipcub::Uninitialized<gpair_sum_t> uninitialized_sum;
  gpair_sum_t& shared_sum = uninitialized_sum.Alias();
  __shared__ ArgMaxT block_max;
  __shared__ TempStorage temp_storage;

  if (threadIdx.x == 0) {
    split = DeviceSplitCandidate();
  }

  __syncthreads();

  // below two are for accessing full-sized node list stored on each device
  // always one block per node, BLOCK_THREADS threads per block
  int level_node_idx = blockIdx.x + nodes_offset_device;
  int node_idx = n_nodes(depth - 1) + level_node_idx;

  for (int fidx = 0; fidx < n_features; fidx++) {
    if (colsample && d_feature_flags[fidx] == 0) continue;

    int begin = d_feature_segments[level_node_idx * n_features + fidx];
    int end = d_feature_segments[level_node_idx * n_features + fidx + 1];

    gpair_sum_t feature_sum = gpair_sum_t();
    for (int reduce_begin = begin; reduce_begin < end;
         reduce_begin += BLOCK_THREADS) {
      bool thread_active = reduce_begin + threadIdx.x < end;
      // Scan histogram
      gpair_sum_t bin = thread_active ? d_level_hist[reduce_begin + threadIdx.x]
                                      : gpair_sum_t();

      feature_sum +=
          SumReduceT(temp_storage.sum_reduce).Reduce(bin, hipcub::Sum());
    }

    if (threadIdx.x == 0) {
      shared_sum = feature_sum;
    }
    //    __syncthreads(); // no need to synch because below there is a Scan

    auto prefix_op = SumCallbackOp<gpair_sum_t>();
    for (int scan_begin = begin; scan_begin < end;
         scan_begin += BLOCK_THREADS) {
      bool thread_active = scan_begin + threadIdx.x < end;
      gpair_sum_t bin = thread_active ? d_level_hist[scan_begin + threadIdx.x]
                                      : gpair_sum_t();

      BlockScanT(temp_storage.scan)
          .ExclusiveScan(bin, bin, hipcub::Sum(), prefix_op);

      // Calculate gain
      gpair_sum_t parent_sum = gpair_sum_t(d_nodes[node_idx].sum_gradients);
      float parent_gain = d_nodes[node_idx].root_gain;

      gpair_sum_t missing = parent_sum - shared_sum;

      bool missing_left;
      float gain = thread_active
                       ? loss_chg_missing(bin, missing, parent_sum, parent_gain,
                                          gpu_param, missing_left)
                       : -FLT_MAX;
      __syncthreads();

      // Find thread with best gain
      ArgMaxT tuple(threadIdx.x, gain);
      ArgMaxT best =
          MaxReduceT(temp_storage.max_reduce).Reduce(tuple, hipcub::ArgMax());

      if (threadIdx.x == 0) {
        block_max = best;
      }

      __syncthreads();

      // Best thread updates split
      if (threadIdx.x == block_max.key) {
        float fvalue;
        int gidx = (scan_begin - (level_node_idx * n_bins)) + threadIdx.x;
        if (threadIdx.x == 0 &&
            begin == scan_begin) {  // check at start of first tile
          fvalue = d_fidx_min_map[fidx];
        } else {
          fvalue = d_gidx_fvalue_map[gidx - 1];
        }

        gpair_sum_t left = missing_left ? bin + missing : bin;
        gpair_sum_t right = parent_sum - left;

        split.Update(gain, missing_left ? LeftDir : RightDir, fvalue, fidx,
                     left, right, gpu_param);
      }
      __syncthreads();
    }  // end scan
  }    // end over features

  // Create node
  if (threadIdx.x == 0 && split.IsValid()) {
    d_nodes[node_idx].SetSplit(split);

    DeviceNodeStats& left_child = d_nodes[left_child_nidx(node_idx)];
    DeviceNodeStats& right_child = d_nodes[right_child_nidx(node_idx)];
    bool& left_child_smallest = d_left_child_smallest_temp[node_idx];
    left_child =
        DeviceNodeStats(split.left_sum, left_child_nidx(node_idx), gpu_param);

    right_child =
        DeviceNodeStats(split.right_sum, right_child_nidx(node_idx), gpu_param);

    // Record smallest node
    if (split.left_sum.GetHess() <= split.right_sum.GetHess()) {
      left_child_smallest = true;
    } else {
      left_child_smallest = false;
    }
  }
}
class GPUHistMaker : public TreeUpdater {
 public:
  GPUHistMaker()
      : initialised(false),
        is_dense(false),
        p_last_fmat_(nullptr),
        prediction_cache_initialised(false) {}
  ~GPUHistMaker() {
    if (initialised) {
      for (int d_idx = 0; d_idx < n_devices; ++d_idx) {
        ncclCommDestroy(comms[d_idx]);

        dh::safe_cuda(hipSetDevice(dList[d_idx]));
        dh::safe_cuda(hipStreamDestroy(*(streams[d_idx])));
      }
      for (int num_d = 1; num_d <= n_devices;
           ++num_d) {  // loop over number of devices used
        for (int d_idx = 0; d_idx < n_devices; ++d_idx) {
          ncclCommDestroy(find_split_comms[num_d - 1][d_idx]);
        }
      }
    }
  }
  void Init(
      const std::vector<std::pair<std::string, std::string>>& args) override {
    param.InitAllowUnknown(args);
    CHECK(param.max_depth < 16) << "Tree depth too large.";
    CHECK(param.max_depth != 0) << "Tree depth cannot be 0.";
    CHECK(param.grow_policy != TrainParam::kLossGuide)
        << "Loss guided growth policy not supported. Use CPU algorithm.";
    this->param = param;

    CHECK(param.n_gpus != 0) << "Must have at least one device";
  }
  void Update(const std::vector<bst_gpair>& gpair, DMatrix* dmat,
              const std::vector<RegTree*>& trees) override {
    GradStats::CheckInfo(dmat->info());
    // rescale learning rate according to size of trees
    float lr = param.learning_rate;
    param.learning_rate = lr / trees.size();
    // build tree
    try {
      for (size_t i = 0; i < trees.size(); ++i) {
        this->UpdateTree(gpair, dmat, trees[i]);
      }
    } catch (const std::exception& e) {
      LOG(FATAL) << "GPU plugin exception: " << e.what() << std::endl;
    }
    param.learning_rate = lr;
  }

  void InitData(const std::vector<bst_gpair>& gpair, DMatrix& fmat,  // NOLINT
                const RegTree& tree) {
    dh::Timer time1;
    // set member num_rows and n_devices for rest of GPUHistBuilder members
    info = &fmat.info();
    num_rows = info->num_row;
    n_devices = dh::n_devices(param.n_gpus, num_rows);

    if (!initialised) {
      // reset static timers used across iterations
      cpu_init_time = 0;
      gpu_init_time = 0;
      cpu_time.Reset();
      gpu_time = 0;

      // set dList member
      dList.resize(n_devices);
      for (int d_idx = 0; d_idx < n_devices; ++d_idx) {
        int device_idx = (param.gpu_id + d_idx) % dh::n_visible_devices();
        dList[d_idx] = device_idx;
      }

      // initialize nccl

      comms.resize(n_devices);
      streams.resize(n_devices);
      dh::safe_nccl(ncclCommInitAll(comms.data(), n_devices,
                                    dList.data()));  // initialize communicator
                                                     // (One communicator per
                                                     // process)

      // printf("# NCCL: Using devices\n");
      for (int d_idx = 0; d_idx < n_devices; ++d_idx) {
        streams[d_idx] =
            reinterpret_cast<hipStream_t*>(malloc(sizeof(hipStream_t)));
        dh::safe_cuda(hipSetDevice(dList[d_idx]));
        dh::safe_cuda(hipStreamCreate(streams[d_idx]));

        int cudaDev;
        int rank;
        hipDeviceProp_t prop;
        dh::safe_nccl(ncclCommCuDevice(comms[d_idx], &cudaDev));
        dh::safe_nccl(ncclCommUserRank(comms[d_idx], &rank));
        dh::safe_cuda(hipGetDeviceProperties(&prop, cudaDev));
        // printf("#   Rank %2d uses device %2d [0x%02x] %s\n", rank, cudaDev,
        //             prop.pciBusID, prop.name);
        // hipDriverGetVersion(&driverVersion);
        // hipRuntimeGetVersion(&runtimeVersion);
        std::ostringstream oss;
        oss << "CUDA Capability Major/Minor version number: " << prop.major
            << "." << prop.minor << " is insufficient.  Need >=3.5.";
        int failed = prop.major < 3 || prop.major == 3 && prop.minor < 5;
        CHECK(failed == 0) << oss.str();
      }

      // local find_split group of comms for each case of reduced number of
      // GPUs to use
      find_split_comms.resize(
          n_devices,
          std::vector<ncclComm_t>(n_devices));  // TODO(JCM): Excessive, but
                                                // ok, and best to do
                                                // here instead of
                                                // repeatedly
      for (int num_d = 1; num_d <= n_devices;
           ++num_d) {  // loop over number of devices used
        dh::safe_nccl(
            ncclCommInitAll(find_split_comms[num_d - 1].data(), num_d,
                            dList.data()));  // initialize communicator
                                             // (One communicator per
                                             // process)
      }

      is_dense = info->num_nonzero == info->num_col * info->num_row;
      dh::Timer time0;
      hmat_.Init(&fmat, param.max_bin);
      cpu_init_time += time0.ElapsedSeconds();
      if (param.debug_verbose) {  // Only done once for each training session
        LOG(CONSOLE) << "[GPU Plug-in] CPU Time for hmat_.Init "
                     << time0.ElapsedSeconds() << " sec";
        fflush(stdout);
      }
      time0.Reset();

      gmat_.cut = &hmat_;
      cpu_init_time += time0.ElapsedSeconds();
      if (param.debug_verbose) {  // Only done once for each training session
        LOG(CONSOLE) << "[GPU Plug-in] CPU Time for gmat_.cut "
                     << time0.ElapsedSeconds() << " sec";
        fflush(stdout);
      }
      time0.Reset();

      gmat_.Init(&fmat);
      cpu_init_time += time0.ElapsedSeconds();
      if (param.debug_verbose) {  // Only done once for each training session
        LOG(CONSOLE) << "[GPU Plug-in] CPU Time for gmat_.Init() "
                     << time0.ElapsedSeconds() << " sec";
        fflush(stdout);
      }
      time0.Reset();

      if (param.debug_verbose) {  // Only done once for each training session
        LOG(CONSOLE)
            << "[GPU Plug-in] CPU Time for hmat_.Init, gmat_.cut, gmat_.Init "
            << cpu_init_time << " sec";
        fflush(stdout);
      }

      int n_bins = hmat_.row_ptr.back();
      int n_features = hmat_.row_ptr.size() - 1;

      // deliniate data onto multiple gpus
      device_row_segments.push_back(0);
      device_element_segments.push_back(0);
      bst_uint offset = 0;
      bst_uint shard_size =
          std::ceil(static_cast<double>(num_rows) / n_devices);
      for (int d_idx = 0; d_idx < n_devices; d_idx++) {
        int device_idx = dList[d_idx];
        offset += shard_size;
        offset = std::min(offset, num_rows);
        device_row_segments.push_back(offset);
        device_element_segments.push_back(gmat_.row_ptr[offset]);
      }

      // Build feature segments
      std::vector<int> h_feature_segments;
      for (int node = 0; node < n_nodes_level(param.max_depth - 1); node++) {
        for (int fidx = 0; fidx < n_features; fidx++) {
          h_feature_segments.push_back(hmat_.row_ptr[fidx] + node * n_bins);
        }
      }
      h_feature_segments.push_back(n_nodes_level(param.max_depth - 1) * n_bins);

      // Construct feature map
      std::vector<int> h_gidx_feature_map(n_bins);
      for (int fidx = 0; fidx < n_features; fidx++) {
        for (int i = hmat_.row_ptr[fidx]; i < hmat_.row_ptr[fidx + 1]; i++) {
          h_gidx_feature_map[i] = fidx;
        }
      }

      int level_max_bins = n_nodes_level(param.max_depth - 1) * n_bins;

      // allocate unique common data that reside on master device (NOTE: None
      // currently)
      //    int master_device=dList[0];
      //    ba.allocate(master_device, );

      // allocate vectors across all devices
      temp_memory.resize(n_devices);
      hist_vec.resize(n_devices);
      nodes.resize(n_devices);
      nodes_temp.resize(n_devices);
      nodes_child_temp.resize(n_devices);
      left_child_smallest.resize(n_devices);
      left_child_smallest_temp.resize(n_devices);
      feature_flags.resize(n_devices);
      fidx_min_map.resize(n_devices);
      feature_segments.resize(n_devices);
      prediction_cache.resize(n_devices);
      position.resize(n_devices);
      position_tmp.resize(n_devices);
      device_matrix.resize(n_devices);
      device_gpair.resize(n_devices);
      gidx_feature_map.resize(n_devices);
      gidx_fvalue_map.resize(n_devices);

      int find_split_n_devices = std::pow(2, std::floor(std::log2(n_devices)));
      find_split_n_devices =
          std::min(n_nodes_level(param.max_depth), find_split_n_devices);
      int max_num_nodes_device =
          n_nodes_level(param.max_depth) / find_split_n_devices;

      // num_rows_segment: for sharding rows onto gpus for splitting data
      // num_elements_segment: for sharding rows (of elements) onto gpus for
      // splitting data
      // max_num_nodes_device: for sharding nodes onto gpus for split finding
      // All other variables have full copy on gpu, with copy either being
      // identical or just current portion (like for histogram) before
      // AllReduce
      for (int d_idx = 0; d_idx < n_devices; d_idx++) {
        int device_idx = dList[d_idx];
        bst_uint num_rows_segment =
            device_row_segments[d_idx + 1] - device_row_segments[d_idx];
        bst_ulong num_elements_segment =
            device_element_segments[d_idx + 1] - device_element_segments[d_idx];
        ba.allocate(
            device_idx, param.silent, &(hist_vec[d_idx].data),
            n_nodes(param.max_depth - 1) * n_bins, &nodes[d_idx],
            n_nodes(param.max_depth), &nodes_temp[d_idx], max_num_nodes_device,
            &nodes_child_temp[d_idx], max_num_nodes_device,
            &left_child_smallest[d_idx], n_nodes(param.max_depth),
            &left_child_smallest_temp[d_idx], max_num_nodes_device,
            &feature_flags[d_idx],
            n_features,  // may change but same on all devices
            &fidx_min_map[d_idx],
            hmat_.min_val.size(),  // constant and same on all devices
            &feature_segments[d_idx],
            h_feature_segments.size(),  // constant and same on all devices
            &prediction_cache[d_idx], num_rows_segment, &position[d_idx],
            num_rows_segment, &position_tmp[d_idx], num_rows_segment,
            &device_gpair[d_idx], num_rows_segment,
            &device_matrix[d_idx].gidx_buffer,
            common::CompressedBufferWriter::CalculateBufferSize(
                num_elements_segment,
                n_bins),  // constant and same on all devices
            &device_matrix[d_idx].row_ptr, num_rows_segment + 1,
            &gidx_feature_map[d_idx],
            n_bins,  // constant and same on all devices
            &gidx_fvalue_map[d_idx],
            hmat_.cut.size());  // constant and same on all devices

        // Copy Host to Device (assumes comes after ba.allocate that sets
        // device)
        device_matrix[d_idx].Init(
            device_idx, gmat_, device_element_segments[d_idx],
            device_element_segments[d_idx + 1], device_row_segments[d_idx],
            device_row_segments[d_idx + 1], n_bins);
        gidx_feature_map[d_idx] = h_gidx_feature_map;
        gidx_fvalue_map[d_idx] = hmat_.cut;
        feature_segments[d_idx] = h_feature_segments;
        fidx_min_map[d_idx] = hmat_.min_val;

        // Initialize, no copy
        hist_vec[d_idx].Init(n_bins);     // init host object
        prediction_cache[d_idx].fill(0);  // init device object (assumes comes
                                          // after ba.allocate that sets device)
        feature_flags[d_idx].fill(
            1);  // init device object (assumes comes after
                 // ba.allocate that sets device)
      }
    }

    // copy or init to do every iteration
    for (int d_idx = 0; d_idx < n_devices; d_idx++) {
      int device_idx = dList[d_idx];
      dh::safe_cuda(hipSetDevice(device_idx));

      nodes[d_idx].fill(DeviceNodeStats());
      nodes_temp[d_idx].fill(DeviceNodeStats());
      nodes_child_temp[d_idx].fill(DeviceNodeStats());

      position[d_idx].fill(0);

      device_gpair[d_idx].copy(gpair.begin() + device_row_segments[d_idx],
                               gpair.begin() + device_row_segments[d_idx + 1]);

      subsample_gpair(&device_gpair[d_idx], param.subsample,
                      device_row_segments[d_idx]);

      hist_vec[d_idx].Reset(device_idx);

      // left_child_smallest and left_child_smallest_temp don't need to be
      // initialized
    }

    dh::synchronize_n_devices(n_devices, dList);

    if (!initialised) {
      gpu_init_time = time1.ElapsedSeconds() - cpu_init_time;
      gpu_time = -cpu_init_time;
      if (param.debug_verbose) {  // Only done once for each training session
        LOG(CONSOLE) << "[GPU Plug-in] Time for GPU operations during First "
                        "Call to InitData() "
                     << gpu_init_time << " sec";
        fflush(stdout);
      }
    }

    p_last_fmat_ = &fmat;

    initialised = true;
  }

  void BuildHist(int depth) {
    for (int d_idx = 0; d_idx < n_devices; d_idx++) {
      int device_idx = dList[d_idx];
      size_t begin = device_element_segments[d_idx];
      size_t end = device_element_segments[d_idx + 1];
      size_t row_begin = device_row_segments[d_idx];
      size_t row_end = device_row_segments[d_idx + 1];

      auto d_gidx = device_matrix[d_idx].gidx;
      auto d_row_ptr = device_matrix[d_idx].row_ptr.tbegin();
      auto d_position = position[d_idx].data();
      auto d_gpair = device_gpair[d_idx].data();
      auto d_left_child_smallest = left_child_smallest[d_idx].data();
      auto hist_builder = hist_vec[d_idx].GetBuilder();
      dh::TransformLbs(
          device_idx, &temp_memory[d_idx], end - begin, d_row_ptr,
          row_end - row_begin, is_dense,
          [=] __device__(size_t local_idx, int local_ridx) {
            int nidx = d_position[local_ridx];  // OPTMARK: latency
            if (!is_active(nidx, depth)) return;

            // Only increment smallest node
            bool is_smallest = (d_left_child_smallest[parent_nidx(nidx)] &&
                                is_left_child(nidx)) ||
                               (!d_left_child_smallest[parent_nidx(nidx)] &&
                                !is_left_child(nidx));
            if (!is_smallest && depth > 0) return;

            int gidx = d_gidx[local_idx];
            bst_gpair gpair = d_gpair[local_ridx];

            hist_builder.Add(gpair, gidx,
                             nidx);  // OPTMARK: This is slow, could use
                                     // shared memory or cache results
                                     // intead of writing to global
                                     // memory every time in atomic way.
          });
    }

    dh::synchronize_n_devices(n_devices, dList);

    //  time.printElapsed("Add Time");

    // (in-place) reduce each element of histogram (for only current level)
    // across multiple gpus
    // TODO(JCM): use out of place with pre-allocated buffer, but then have to
    // copy
    // back on device
    //  fprintf(stderr,"sizeof(bst_gpair)/sizeof(float)=%d\n",sizeof(bst_gpair)/sizeof(float));
    for (int d_idx = 0; d_idx < n_devices; d_idx++) {
      int device_idx = dList[d_idx];
      dh::safe_cuda(hipSetDevice(device_idx));
      dh::safe_nccl(ncclAllReduce(
          reinterpret_cast<const void*>(hist_vec[d_idx].GetLevelPtr(depth)),
          reinterpret_cast<void*>(hist_vec[d_idx].GetLevelPtr(depth)),
          hist_vec[d_idx].LevelSize(depth) * sizeof(gpair_sum_t) /
              sizeof(gpair_sum_t::value_t),
          nccl_sum_t, ncclSum, comms[d_idx], *(streams[d_idx])));
    }

    for (int d_idx = 0; d_idx < n_devices; d_idx++) {
      int device_idx = dList[d_idx];
      dh::safe_cuda(hipSetDevice(device_idx));
      dh::safe_cuda(hipStreamSynchronize(*(streams[d_idx])));
    }
    // if no NCCL, then presume only 1 GPU, then already correct

    //  time.printElapsed("Reduce-Add Time");

    // Subtraction trick (applied to all devices in same way -- to avoid doing
    // on master and then Bcast)
    if (depth > 0) {
      for (int d_idx = 0; d_idx < n_devices; d_idx++) {
        int device_idx = dList[d_idx];
        dh::safe_cuda(hipSetDevice(device_idx));

        auto hist_builder = hist_vec[d_idx].GetBuilder();
        auto d_left_child_smallest = left_child_smallest[d_idx].data();
        int n_sub_bins = (n_nodes_level(depth) / 2) * hist_builder.n_bins;

        dh::launch_n(device_idx, n_sub_bins, [=] __device__(int idx) {
          int nidx = n_nodes(depth - 1) + ((idx / hist_builder.n_bins) * 2);
          bool left_smallest = d_left_child_smallest[parent_nidx(nidx)];
          if (left_smallest) {
            nidx++;  // If left is smallest switch to right child
          }

          int gidx = idx % hist_builder.n_bins;
          gpair_sum_t parent = hist_builder.Get(gidx, parent_nidx(nidx));
          int other_nidx = left_smallest ? nidx - 1 : nidx + 1;
          gpair_sum_t other = hist_builder.Get(gidx, other_nidx);
          gpair_sum_t sub = parent - other;
          hist_builder.Add(
              bst_gpair(sub.GetGrad(), sub.GetHess()), gidx,
              nidx);  // OPTMARK: This is slow, could use shared
                      // memory or cache results intead of writing to
                      // global memory every time in atomic way.
        });
      }
      dh::synchronize_n_devices(n_devices, dList);
    }
  }
#define MIN_BLOCK_THREADS 128
#define CHUNK_BLOCK_THREADS 128
// MAX_BLOCK_THREADS of 1024 is hard-coded maximum block size due
// to CUDA capability 35 and above requirement
// for Maximum number of threads per block
#define MAX_BLOCK_THREADS 512

  void FindSplit(int depth) {
    // Specialised based on max_bins
    this->FindSplitSpecialize(depth, Int<MIN_BLOCK_THREADS>());
  }

  template <int BLOCK_THREADS>
  void FindSplitSpecialize(int depth, Int<BLOCK_THREADS>) {
    if (param.max_bin <= BLOCK_THREADS) {
      LaunchFindSplit<BLOCK_THREADS>(depth);
    } else {
      this->FindSplitSpecialize(depth,
                                Int<BLOCK_THREADS + CHUNK_BLOCK_THREADS>());
    }
  }

  void FindSplitSpecialize(int depth, Int<MAX_BLOCK_THREADS>) {
    this->LaunchFindSplit<MAX_BLOCK_THREADS>(depth);
  }

  template <int BLOCK_THREADS>
  void LaunchFindSplit(int depth) {
    bool colsample =
        param.colsample_bylevel < 1.0 || param.colsample_bytree < 1.0;

    int num_nodes_device = n_nodes_level(depth);
    const int GRID_SIZE = num_nodes_device;

    // all GPUs do same work
    for (int d_idx = 0; d_idx < n_devices; d_idx++) {
      int device_idx = dList[d_idx];
      dh::safe_cuda(hipSetDevice(device_idx));

      int nodes_offset_device = 0;
      find_split_kernel<BLOCK_THREADS><<<GRID_SIZE, BLOCK_THREADS>>>(
          hist_vec[d_idx].GetLevelPtr(depth), feature_segments[d_idx].data(),
          depth, (info->num_col), (hmat_.row_ptr.back()), nodes[d_idx].data(),
          nodes_offset_device, fidx_min_map[d_idx].data(),
          gidx_fvalue_map[d_idx].data(), GPUTrainingParam(param),
          left_child_smallest[d_idx].data(), colsample,
          feature_flags[d_idx].data());
    }

    // NOTE: No need to syncrhonize with host as all above pure P2P ops or
    // on-device ops
  }
  void InitFirstNode(const std::vector<bst_gpair>& gpair) {
    // Perform asynchronous reduction on each gpu
    std::vector<bst_gpair> device_sums(n_devices);
#pragma omp parallel for num_threads(n_devices)
    for (int d_idx = 0; d_idx < n_devices; d_idx++) {
      int device_idx = dList[d_idx];
      dh::safe_cuda(hipSetDevice(device_idx));
      auto begin = device_gpair[d_idx].tbegin();
      auto end = device_gpair[d_idx].tend();
      bst_gpair init = bst_gpair();
      auto binary_op = thrust::plus<bst_gpair>();
      device_sums[d_idx] = thrust::reduce(begin, end, init, binary_op);
    }

    bst_gpair sum = bst_gpair();
    for (int d_idx = 0; d_idx < n_devices; d_idx++) {
      sum += device_sums[d_idx];
    }

    // Setup first node so all devices have same first node (here done same on
    // all devices, or could have done one device and Bcast if worried about
    // exact precision issues)
    for (int d_idx = 0; d_idx < n_devices; d_idx++) {
      int device_idx = dList[d_idx];

      auto d_nodes = nodes[d_idx].data();
      auto gpu_param = GPUTrainingParam(param);

      dh::launch_n(device_idx, 1, [=] __device__(int idx) {
        bst_gpair sum_gradients = sum;
        d_nodes[idx] = DeviceNodeStats(sum_gradients, 0, gpu_param);
      });
    }
    // synch all devices to host before moving on (No, can avoid because
    // BuildHist calls another kernel in default stream)
    //  dh::synchronize_n_devices(n_devices, dList);
  }
  void UpdatePosition(int depth) {
    if (is_dense) {
      this->UpdatePositionDense(depth);
    } else {
      this->UpdatePositionSparse(depth);
    }
  }
  void UpdatePositionDense(int depth) {
    for (int d_idx = 0; d_idx < n_devices; d_idx++) {
      int device_idx = dList[d_idx];

      auto d_position = position[d_idx].data();
      DeviceNodeStats* d_nodes = nodes[d_idx].data();
      auto d_gidx_fvalue_map = gidx_fvalue_map[d_idx].data();
      auto d_gidx = device_matrix[d_idx].gidx;
      int n_columns = info->num_col;
      size_t begin = device_row_segments[d_idx];
      size_t end = device_row_segments[d_idx + 1];

      dh::launch_n(device_idx, end - begin, [=] __device__(size_t local_idx) {
        int pos = d_position[local_idx];
        if (!is_active(pos, depth)) {
          return;
        }
        DeviceNodeStats node = d_nodes[pos];

        if (node.IsLeaf()) {
          return;
        }

        int gidx = d_gidx[local_idx * static_cast<size_t>(n_columns) +
                          static_cast<size_t>(node.fidx)];

        float fvalue = d_gidx_fvalue_map[gidx];

        if (fvalue <= node.fvalue) {
          d_position[local_idx] = left_child_nidx(pos);
        } else {
          d_position[local_idx] = right_child_nidx(pos);
        }
      });
    }
    dh::synchronize_n_devices(n_devices, dList);
    // dh::safe_cuda(hipDeviceSynchronize());
  }

  void UpdatePositionSparse(int depth) {
    for (int d_idx = 0; d_idx < n_devices; d_idx++) {
      int device_idx = dList[d_idx];

      auto d_position = position[d_idx].data();
      auto d_position_tmp = position_tmp[d_idx].data();
      DeviceNodeStats* d_nodes = nodes[d_idx].data();
      auto d_gidx_feature_map = gidx_feature_map[d_idx].data();
      auto d_gidx_fvalue_map = gidx_fvalue_map[d_idx].data();
      auto d_gidx = device_matrix[d_idx].gidx;
      auto d_row_ptr = device_matrix[d_idx].row_ptr.tbegin();

      size_t row_begin = device_row_segments[d_idx];
      size_t row_end = device_row_segments[d_idx + 1];
      size_t element_begin = device_element_segments[d_idx];
      size_t element_end = device_element_segments[d_idx + 1];

      // Update missing direction
      dh::launch_n(device_idx, row_end - row_begin,
                   [=] __device__(int local_idx) {
                     int pos = d_position[local_idx];
                     if (!is_active(pos, depth)) {
                       d_position_tmp[local_idx] = pos;
                       return;
                     }

                     DeviceNodeStats node = d_nodes[pos];

                     if (node.IsLeaf()) {
                       d_position_tmp[local_idx] = pos;
                       return;
                     } else if (node.dir == LeftDir) {
                       d_position_tmp[local_idx] = pos * 2 + 1;
                     } else {
                       d_position_tmp[local_idx] = pos * 2 + 2;
                     }
                   });

      // Update node based on fvalue where exists
      // OPTMARK: This kernel is very inefficient for both compute and memory,
      // dominated by memory dependency / access patterns

      dh::TransformLbs(
          device_idx, &temp_memory[d_idx], element_end - element_begin,
          d_row_ptr, row_end - row_begin, is_dense,
          [=] __device__(size_t local_idx, int local_ridx) {
            int pos = d_position[local_ridx];
            if (!is_active(pos, depth)) {
              return;
            }

            DeviceNodeStats node = d_nodes[pos];

            if (node.IsLeaf()) {
              return;
            }

            int gidx = d_gidx[local_idx];
            int findex =
                d_gidx_feature_map[gidx];  // OPTMARK: slowest global
                                           // memory access, maybe setup
                                           // position, gidx, etc. as
                                           // combined structure?

            if (findex == node.fidx) {
              float fvalue = d_gidx_fvalue_map[gidx];

              if (fvalue <= node.fvalue) {
                d_position_tmp[local_ridx] = left_child_nidx(pos);
              } else {
                d_position_tmp[local_ridx] = right_child_nidx(pos);
              }
            }
          });
      position[d_idx] = position_tmp[d_idx];
    }
    dh::synchronize_n_devices(n_devices, dList);
  }
  void ColSampleTree() {
    if (param.colsample_bylevel == 1.0 && param.colsample_bytree == 1.0) return;

    feature_set_tree.resize(info->num_col);
    std::iota(feature_set_tree.begin(), feature_set_tree.end(), 0);
    feature_set_tree = col_sample(feature_set_tree, param.colsample_bytree);
  }
  void ColSampleLevel() {
    if (param.colsample_bylevel == 1.0 && param.colsample_bytree == 1.0) return;

    feature_set_level.resize(feature_set_tree.size());
    feature_set_level = col_sample(feature_set_tree, param.colsample_bylevel);
    std::vector<int> h_feature_flags(info->num_col, 0);
    for (auto fidx : feature_set_level) {
      h_feature_flags[fidx] = 1;
    }

    for (int d_idx = 0; d_idx < n_devices; d_idx++) {
      int device_idx = dList[d_idx];
      dh::safe_cuda(hipSetDevice(device_idx));

      feature_flags[d_idx] = h_feature_flags;
    }
    dh::synchronize_n_devices(n_devices, dList);
  }
  bool UpdatePredictionCache(const DMatrix* data,
                             std::vector<bst_float>* p_out_preds) override {
    std::vector<bst_float>& out_preds = *p_out_preds;

    if (nodes.empty() || !p_last_fmat_ || data != p_last_fmat_) {
      return false;
    }

    if (!prediction_cache_initialised) {
      for (int d_idx = 0; d_idx < n_devices; d_idx++) {
        int device_idx = dList[d_idx];
        size_t row_begin = device_row_segments[d_idx];
        size_t row_end = device_row_segments[d_idx + 1];

        prediction_cache[d_idx].copy(out_preds.begin() + row_begin,
                                     out_preds.begin() + row_end);
      }
      prediction_cache_initialised = true;
    }
    dh::synchronize_n_devices(n_devices, dList);

    float eps = param.learning_rate;
    for (int d_idx = 0; d_idx < n_devices; d_idx++) {
      int device_idx = dList[d_idx];
      size_t row_begin = device_row_segments[d_idx];
      size_t row_end = device_row_segments[d_idx + 1];

      auto d_nodes = nodes[d_idx].data();
      auto d_position = position[d_idx].data();
      auto d_prediction_cache = prediction_cache[d_idx].data();

      dh::launch_n(device_idx, prediction_cache[d_idx].size(),
                   [=] __device__(int local_idx) {
                     int pos = d_position[local_idx];
                     d_prediction_cache[local_idx] += d_nodes[pos].weight * eps;
                   });

      dh::safe_cuda(
          hipMemcpy(&out_preds[row_begin], prediction_cache[d_idx].data(),
                     prediction_cache[d_idx].size() * sizeof(bst_float),
                     hipMemcpyDeviceToHost));
    }
    dh::synchronize_n_devices(n_devices, dList);

    return true;
  }
  void UpdateTree(const std::vector<bst_gpair>& gpair, DMatrix* p_fmat,
                  RegTree* p_tree) {
    dh::Timer time0;

    this->InitData(gpair, *p_fmat, *p_tree);
    this->InitFirstNode(gpair);
    this->ColSampleTree();

    for (int depth = 0; depth < param.max_depth; depth++) {
      this->ColSampleLevel();
      this->BuildHist(depth);
      this->FindSplit(depth);
      this->UpdatePosition(depth);
    }

    // done with multi-GPU, pass back result from master to tree on host
    int master_device = dList[0];
    dh::safe_cuda(hipSetDevice(master_device));
    dense2sparse_tree(p_tree, nodes[0], param);

    gpu_time += time0.ElapsedSeconds();

    if (param.debug_verbose) {
      LOG(CONSOLE)
          << "[GPU Plug-in] Cumulative GPU Time excluding initial time "
          << (gpu_time - gpu_init_time) << " sec";
      fflush(stdout);
    }

    if (param.debug_verbose) {
      LOG(CONSOLE) << "[GPU Plug-in] Cumulative CPU Time "
                   << cpu_time.ElapsedSeconds() << " sec";
      LOG(CONSOLE)
          << "[GPU Plug-in] Cumulative CPU Time excluding initial time "
          << (cpu_time.ElapsedSeconds() - cpu_init_time - gpu_time) << " sec";
      fflush(stdout);
    }
  }

 protected:
  TrainParam param;
  // std::unique_ptr<GPUHistBuilder> builder;
  common::HistCutMatrix hmat_;
  common::GHistIndexMatrix gmat_;
  MetaInfo* info;
  bool initialised;
  bool is_dense;
  const DMatrix* p_last_fmat_;
  bool prediction_cache_initialised;

  dh::bulk_allocator<dh::memory_type::DEVICE> ba;

  std::vector<int> feature_set_tree;
  std::vector<int> feature_set_level;

  bst_uint num_rows;
  int n_devices;

  // below vectors are for each devices used
  std::vector<int> dList;
  std::vector<int> device_row_segments;
  std::vector<size_t> device_element_segments;

  std::vector<dh::CubMemory> temp_memory;
  std::vector<DeviceHist> hist_vec;
  std::vector<dh::dvec<DeviceNodeStats>> nodes;
  std::vector<dh::dvec<DeviceNodeStats>> nodes_temp;
  std::vector<dh::dvec<DeviceNodeStats>> nodes_child_temp;
  std::vector<dh::dvec<bool>> left_child_smallest;
  std::vector<dh::dvec<bool>> left_child_smallest_temp;
  std::vector<dh::dvec<int>> feature_flags;
  std::vector<dh::dvec<float>> fidx_min_map;
  std::vector<dh::dvec<int>> feature_segments;
  std::vector<dh::dvec<bst_float>> prediction_cache;
  std::vector<dh::dvec<int>> position;
  std::vector<dh::dvec<int>> position_tmp;
  std::vector<DeviceGMat> device_matrix;
  std::vector<dh::dvec<bst_gpair>> device_gpair;
  std::vector<dh::dvec<int>> gidx_feature_map;
  std::vector<dh::dvec<float>> gidx_fvalue_map;

  std::vector<hipStream_t*> streams;
  std::vector<ncclComm_t> comms;
  std::vector<std::vector<ncclComm_t>> find_split_comms;

  double cpu_init_time;
  double gpu_init_time;
  dh::Timer cpu_time;
  double gpu_time;
};

XGBOOST_REGISTER_TREE_UPDATER(GPUHistMaker, "grow_gpu_hist")
    .describe("Grow tree with GPU.")
    .set_body([]() { return new GPUHistMaker(); });
}  // namespace tree
}  // namespace xgboost
