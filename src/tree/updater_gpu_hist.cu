#include "hip/hip_runtime.h"
/**
 * Copyright 2017-2023 by XGBoost contributors
 */
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <xgboost/tree_updater.h>

#include <algorithm>
#include <cmath>
#include <cstddef>  // for size_t
#include <memory>   // for unique_ptr, make_unique
#include <utility>  // for move
#include <vector>

#include "../collective/communicator-inl.cuh"
#include "../common/bitfield.h"
#include "../common/categorical.h"
#include "../common/cuda_context.cuh"  // HIPContext
#include "../common/device_helpers.cuh"
#include "../common/hist_util.h"
#include "../common/io.h"
#include "../common/timer.h"
#include "../data/ellpack_page.cuh"
#include "../data/ellpack_page.h"
#include "constraints.cuh"
#include "driver.h"
#include "gpu_hist/evaluate_splits.cuh"
#include "gpu_hist/expand_entry.cuh"
#include "gpu_hist/feature_groups.cuh"
#include "gpu_hist/gradient_based_sampler.cuh"
#include "gpu_hist/histogram.cuh"
#include "gpu_hist/row_partitioner.cuh"
#include "hist/param.h"
#include "param.h"
#include "updater_gpu_common.cuh"
#include "xgboost/base.h"
#include "xgboost/context.h"
#include "xgboost/data.h"
#include "xgboost/host_device_vector.h"
#include "xgboost/json.h"
#include "xgboost/parameter.h"
#include "xgboost/span.h"
#include "xgboost/task.h"  // for ObjInfo
#include "xgboost/tree_model.h"

namespace xgboost::tree {
#if !defined(GTEST_TEST)
DMLC_REGISTRY_FILE_TAG(updater_gpu_hist);
#endif  // !defined(GTEST_TEST)

/**
 * \struct  DeviceHistogramStorage
 *
 * \summary Data storage for node histograms on device. Automatically expands.
 *
 * \tparam GradientSumT      histogram entry type.
 * \tparam kStopGrowingSize  Do not grow beyond this size
 *
 * \author  Rory
 * \date    28/07/2018
 */
template <size_t kStopGrowingSize = 1 << 28>
class DeviceHistogramStorage {
 private:
  using GradientSumT = GradientPairInt64;
  /*! \brief Map nidx to starting index of its histogram. */
  std::map<int, size_t> nidx_map_;
  // Large buffer of zeroed memory, caches histograms
  dh::device_vector<typename GradientSumT::ValueT> data_;
  // If we run out of storage allocate one histogram at a time
  // in overflow. Not cached, overwritten when a new histogram
  // is requested
  dh::device_vector<typename GradientSumT::ValueT> overflow_;
  std::map<int, size_t> overflow_nidx_map_;
  int n_bins_;
  int device_id_;
  static constexpr size_t kNumItemsInGradientSum =
      sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT);
  static_assert(kNumItemsInGradientSum == 2, "Number of items in gradient type should be 2.");

 public:
  // Start with about 16mb
  DeviceHistogramStorage() { data_.reserve(1 << 22); }
  void Init(int device_id, int n_bins) {
    this->n_bins_ = n_bins;
    this->device_id_ = device_id;
  }

  void Reset() {
    auto d_data = data_.data().get();
    dh::LaunchN(data_.size(), [=] __device__(size_t idx) { d_data[idx] = 0.0f; });
    nidx_map_.clear();
    overflow_nidx_map_.clear();
  }
  [[nodiscard]] bool HistogramExists(int nidx) const {
    return nidx_map_.find(nidx) != nidx_map_.cend() ||
           overflow_nidx_map_.find(nidx) != overflow_nidx_map_.cend();
  }
  [[nodiscard]] int Bins() const { return n_bins_; }
  [[nodiscard]] size_t HistogramSize() const { return n_bins_ * kNumItemsInGradientSum; }
  dh::device_vector<typename GradientSumT::ValueT>& Data() { return data_; }

  void AllocateHistograms(const std::vector<int>& new_nidxs) {
    for (int nidx : new_nidxs) {
      CHECK(!HistogramExists(nidx));
    }
    // Number of items currently used in data
    const size_t used_size = nidx_map_.size() * HistogramSize();
    const size_t new_used_size = used_size + HistogramSize() * new_nidxs.size();
    if (used_size >= kStopGrowingSize) {
      // Use overflow
      // Delete previous entries
      overflow_nidx_map_.clear();
      overflow_.resize(HistogramSize() * new_nidxs.size());
      // Zero memory
      auto d_data = overflow_.data().get();
      dh::LaunchN(overflow_.size(),
                  [=] __device__(size_t idx) { d_data[idx] = 0.0; });
      // Append new histograms
      for (int nidx : new_nidxs) {
        overflow_nidx_map_[nidx] = overflow_nidx_map_.size() * HistogramSize();
      }
    } else {
      CHECK_GE(data_.size(), used_size);
      // Expand if necessary
      if (data_.size() < new_used_size) {
        data_.resize(std::max(data_.size() * 2, new_used_size));
      }
      // Append new histograms
      for (int nidx : new_nidxs) {
        nidx_map_[nidx] = nidx_map_.size() * HistogramSize();
      }
    }

    CHECK_GE(data_.size(), nidx_map_.size() * HistogramSize());
  }

  /**
   * \summary   Return pointer to histogram memory for a given node.
   * \param nidx    Tree node index.
   * \return    hist pointer.
   */
  common::Span<GradientSumT> GetNodeHistogram(int nidx) {
    CHECK(this->HistogramExists(nidx));

    if (nidx_map_.find(nidx) != nidx_map_.cend()) {
      // Fetch from normal cache
      auto ptr = data_.data().get() + nidx_map_.at(nidx);
      return {reinterpret_cast<GradientSumT*>(ptr), static_cast<std::size_t>(n_bins_)};
    } else {
      // Fetch from overflow
      auto ptr = overflow_.data().get() + overflow_nidx_map_.at(nidx);
      return {reinterpret_cast<GradientSumT*>(ptr), static_cast<std::size_t>(n_bins_)};
    }
  }
};

// Manage memory for a single GPU
struct GPUHistMakerDevice {
 private:
  GPUHistEvaluator evaluator_;
  Context const* ctx_;
  std::shared_ptr<common::ColumnSampler> column_sampler_;

 public:
  EllpackPageImpl const* page{nullptr};
  common::Span<FeatureType const> feature_types;

  std::unique_ptr<RowPartitioner> row_partitioner;
  DeviceHistogramStorage<> hist{};

  dh::device_vector<GradientPair> d_gpair;  // storage for gpair;
  common::Span<GradientPair> gpair;

  dh::device_vector<int> monotone_constraints;
  // node idx for each sample
  dh::device_vector<bst_node_t> positions;

  TrainParam param;

  std::unique_ptr<GradientQuantiser> quantiser;

  dh::PinnedMemory pinned;
  dh::PinnedMemory pinned2;

  common::Monitor monitor;
  FeatureInteractionConstraintDevice interaction_constraints;

  std::unique_ptr<GradientBasedSampler> sampler;

  std::unique_ptr<FeatureGroups> feature_groups;

  GPUHistMakerDevice(Context const* ctx, bool is_external_memory,
                     common::Span<FeatureType const> _feature_types, bst_row_t _n_rows,
                     TrainParam _param, std::shared_ptr<common::ColumnSampler> column_sampler,
                     uint32_t n_features, BatchParam batch_param)
      : evaluator_{_param, n_features, ctx->gpu_id},
        ctx_(ctx),
        feature_types{_feature_types},
        param(std::move(_param)),
        column_sampler_(std::move(column_sampler)),
        interaction_constraints(param, n_features) {
    sampler = std::make_unique<GradientBasedSampler>(ctx, _n_rows, batch_param, param.subsample,
                                                     param.sampling_method, is_external_memory);
    if (!param.monotone_constraints.empty()) {
      // Copy assigning an empty vector causes an exception in MSVC debug builds
      monotone_constraints = param.monotone_constraints;
    }

    CHECK(column_sampler_);
    monitor.Init(std::string("GPUHistMakerDevice") + std::to_string(ctx_->gpu_id));
  }

  ~GPUHistMakerDevice() = default;

  void InitFeatureGroupsOnce() {
    if (!feature_groups) {
      CHECK(page);
      feature_groups = std::make_unique<FeatureGroups>(page->Cuts(), page->is_dense,
                                                       dh::MaxSharedMemoryOptin(ctx_->gpu_id),
                                                       sizeof(GradientPairPrecise));
    }
  }

  // Reset values for each update iteration
  void Reset(HostDeviceVector<GradientPair>* dh_gpair, DMatrix* dmat, int64_t num_columns) {
    auto const& info = dmat->Info();
    this->column_sampler_->Init(ctx_, num_columns, info.feature_weights.HostVector(),
                                param.colsample_bynode, param.colsample_bylevel,
                                param.colsample_bytree);
    dh::safe_cuda(hipSetDevice(ctx_->gpu_id));

    this->interaction_constraints.Reset();

    if (d_gpair.size() != dh_gpair->Size()) {
      d_gpair.resize(dh_gpair->Size());
    }
    dh::safe_cuda(hipMemcpyAsync(d_gpair.data().get(), dh_gpair->ConstDevicePointer(),
                                  dh_gpair->Size() * sizeof(GradientPair),
                                  hipMemcpyDeviceToDevice));
    auto sample = sampler->Sample(ctx_, dh::ToSpan(d_gpair), dmat);
    page = sample.page;
    gpair = sample.gpair;

    this->evaluator_.Reset(page->Cuts(), feature_types, dmat->Info().num_col_, param, ctx_->gpu_id);

    quantiser = std::make_unique<GradientQuantiser>(this->gpair);

    row_partitioner.reset();  // Release the device memory first before reallocating
    row_partitioner = std::make_unique<RowPartitioner>(ctx_->gpu_id, sample.sample_rows);

    // Init histogram
    hist.Init(ctx_->gpu_id, page->Cuts().TotalBins());
    hist.Reset();

    this->InitFeatureGroupsOnce();
  }

  GPUExpandEntry EvaluateRootSplit(GradientPairInt64 root_sum) {
    int nidx = RegTree::kRoot;
    GPUTrainingParam gpu_param(param);
    auto sampled_features = column_sampler_->GetFeatureSet(0);
    sampled_features->SetDevice(ctx_->Device());
    common::Span<bst_feature_t> feature_set =
        interaction_constraints.Query(sampled_features->DeviceSpan(), nidx);
    auto matrix = page->GetDeviceAccessor(ctx_->gpu_id);
    EvaluateSplitInputs inputs{nidx, 0, root_sum, feature_set, hist.GetNodeHistogram(nidx)};
    EvaluateSplitSharedInputs shared_inputs{
        gpu_param,
        *quantiser,
        feature_types,
        matrix.feature_segments,
        matrix.gidx_fvalue_map,
        matrix.min_fvalue,
        matrix.is_dense && !collective::IsDistributed()
    };
    auto split = this->evaluator_.EvaluateSingleSplit(inputs, shared_inputs);
    return split;
  }

  void EvaluateSplits(const std::vector<GPUExpandEntry>& candidates, const RegTree& tree,
                               common::Span<GPUExpandEntry> pinned_candidates_out) {
    if (candidates.empty()) return;
    dh::TemporaryArray<EvaluateSplitInputs> d_node_inputs(2 * candidates.size());
    dh::TemporaryArray<DeviceSplitCandidate> splits_out(2 * candidates.size());
    std::vector<bst_node_t> nidx(2 * candidates.size());
    auto h_node_inputs = pinned2.GetSpan<EvaluateSplitInputs>(2 * candidates.size());
    auto matrix = page->GetDeviceAccessor(ctx_->gpu_id);
    EvaluateSplitSharedInputs shared_inputs{GPUTrainingParam{param}, *quantiser, feature_types,
                                            matrix.feature_segments, matrix.gidx_fvalue_map,
                                            matrix.min_fvalue,
                                            // is_dense represents the local data
                                            matrix.is_dense && !collective::IsDistributed()};
    dh::TemporaryArray<GPUExpandEntry> entries(2 * candidates.size());
    // Store the feature set ptrs so they dont go out of scope before the kernel is called
    std::vector<std::shared_ptr<HostDeviceVector<bst_feature_t>>> feature_sets;
    for (std::size_t i = 0; i < candidates.size(); i++) {
      auto candidate = candidates.at(i);
      int left_nidx = tree[candidate.nid].LeftChild();
      int right_nidx = tree[candidate.nid].RightChild();
      nidx[i * 2] = left_nidx;
      nidx[i * 2 + 1] = right_nidx;
      auto left_sampled_features = column_sampler_->GetFeatureSet(tree.GetDepth(left_nidx));
      left_sampled_features->SetDevice(ctx_->Device());
      feature_sets.emplace_back(left_sampled_features);
      common::Span<bst_feature_t> left_feature_set =
          interaction_constraints.Query(left_sampled_features->DeviceSpan(), left_nidx);
      auto right_sampled_features = column_sampler_->GetFeatureSet(tree.GetDepth(right_nidx));
      right_sampled_features->SetDevice(ctx_->Device());
      feature_sets.emplace_back(right_sampled_features);
      common::Span<bst_feature_t> right_feature_set =
          interaction_constraints.Query(right_sampled_features->DeviceSpan(),
                                        right_nidx);
      h_node_inputs[i * 2] = {left_nidx, candidate.depth + 1,
                              candidate.split.left_sum, left_feature_set,
                              hist.GetNodeHistogram(left_nidx)};
      h_node_inputs[i * 2 + 1] = {right_nidx, candidate.depth + 1,
                                  candidate.split.right_sum, right_feature_set,
                                  hist.GetNodeHistogram(right_nidx)};
    }
    bst_feature_t max_active_features = 0;
    for (auto input : h_node_inputs) {
      max_active_features =
          std::max(max_active_features, static_cast<bst_feature_t>(input.feature_set.size()));
    }
    dh::safe_cuda(hipMemcpyAsync(
        d_node_inputs.data().get(), h_node_inputs.data(),
        h_node_inputs.size() * sizeof(EvaluateSplitInputs), hipMemcpyDefault));

    this->evaluator_.EvaluateSplits(nidx, max_active_features, dh::ToSpan(d_node_inputs),
                                    shared_inputs, dh::ToSpan(entries));
    dh::safe_cuda(hipMemcpyAsync(pinned_candidates_out.data(),
                                  entries.data().get(), sizeof(GPUExpandEntry) * entries.size(),
                                  hipMemcpyDeviceToHost));
    dh::DefaultStream().Sync();
  }

  void BuildHist(int nidx) {
    auto d_node_hist = hist.GetNodeHistogram(nidx);
    auto d_ridx = row_partitioner->GetRows(nidx);
    BuildGradientHistogram(ctx_->CUDACtx(), page->GetDeviceAccessor(ctx_->gpu_id),
                           feature_groups->DeviceAccessor(ctx_->gpu_id), gpair, d_ridx, d_node_hist,
                           *quantiser);
  }

  // Attempt to do subtraction trick
  // return true if succeeded
  bool SubtractionTrick(int nidx_parent, int nidx_histogram, int nidx_subtraction) {
    if (!hist.HistogramExists(nidx_histogram) || !hist.HistogramExists(nidx_parent)) {
      return false;
    }
    auto d_node_hist_parent = hist.GetNodeHistogram(nidx_parent);
    auto d_node_hist_histogram = hist.GetNodeHistogram(nidx_histogram);
    auto d_node_hist_subtraction = hist.GetNodeHistogram(nidx_subtraction);

    dh::LaunchN(page->Cuts().TotalBins(), [=] __device__(size_t idx) {
      d_node_hist_subtraction[idx] =
          d_node_hist_parent[idx] - d_node_hist_histogram[idx];
    });
    return true;
  }

  // Extra data for each node that is passed
  // to the update position function
  struct NodeSplitData {
    RegTree::Node split_node;
    FeatureType split_type;
    common::KCatBitField node_cats;
  };

  void UpdatePosition(std::vector<GPUExpandEntry> const& candidates, RegTree* p_tree) {
    if (candidates.empty()) {
      return;
    }

    std::vector<bst_node_t> nidx(candidates.size());
    std::vector<bst_node_t> left_nidx(candidates.size());
    std::vector<bst_node_t> right_nidx(candidates.size());
    std::vector<NodeSplitData> split_data(candidates.size());

    for (size_t i = 0; i < candidates.size(); i++) {
      auto const& e = candidates[i];
      RegTree::Node split_node = (*p_tree)[e.nid];
      auto split_type = p_tree->NodeSplitType(e.nid);
      nidx.at(i) = e.nid;
      left_nidx.at(i) = split_node.LeftChild();
      right_nidx.at(i) = split_node.RightChild();
      split_data.at(i) = NodeSplitData{split_node, split_type, evaluator_.GetDeviceNodeCats(e.nid)};

      CHECK_EQ(split_type == FeatureType::kCategorical, e.split.is_cat);
    }

    auto d_matrix = page->GetDeviceAccessor(ctx_->gpu_id);
    row_partitioner->UpdatePositionBatch(
        nidx, left_nidx, right_nidx, split_data,
        [=] __device__(bst_uint ridx, const NodeSplitData& data) {
          // given a row index, returns the node id it belongs to
          float cut_value = d_matrix.GetFvalue(ridx, data.split_node.SplitIndex());
          // Missing value
          bool go_left = true;
          if (isnan(cut_value)) {
            go_left = data.split_node.DefaultLeft();
          } else {
            if (data.split_type == FeatureType::kCategorical) {
              go_left = common::Decision(data.node_cats.Bits(), cut_value);
            } else {
              go_left = cut_value <= data.split_node.SplitCond();
            }
          }
          return go_left;
        });
  }

  // After tree update is finished, update the position of all training
  // instances to their final leaf. This information is used later to update the
  // prediction cache
  void FinalisePosition(RegTree const* p_tree, DMatrix* p_fmat, ObjInfo task,
                        HostDeviceVector<bst_node_t>* p_out_position) {
    // Prediction cache will not be used with external memory
    if (!p_fmat->SingleColBlock()) {
      if (task.UpdateTreeLeaf()) {
        LOG(FATAL) << "Current objective function can not be used with external memory.";
      }
      p_out_position->Resize(0);
      positions.clear();
      return;
    }

    dh::TemporaryArray<RegTree::Node> d_nodes(p_tree->GetNodes().size());
    dh::safe_cuda(hipMemcpyAsync(d_nodes.data().get(), p_tree->GetNodes().data(),
                                  d_nodes.size() * sizeof(RegTree::Node),
                                  hipMemcpyHostToDevice));
    auto const& h_split_types = p_tree->GetSplitTypes();
    auto const& categories = p_tree->GetSplitCategories();
    auto const& categories_segments = p_tree->GetSplitCategoriesPtr();

    dh::caching_device_vector<FeatureType> d_split_types;
    dh::caching_device_vector<uint32_t> d_categories;
    dh::caching_device_vector<RegTree::CategoricalSplitMatrix::Segment> d_categories_segments;

    if (!categories.empty()) {
      dh::CopyToD(h_split_types, &d_split_types);
      dh::CopyToD(categories, &d_categories);
      dh::CopyToD(categories_segments, &d_categories_segments);
    }

    FinalisePositionInPage(page, dh::ToSpan(d_nodes), dh::ToSpan(d_split_types),
                           dh::ToSpan(d_categories), dh::ToSpan(d_categories_segments),
                           p_out_position);
  }

  void FinalisePositionInPage(
      EllpackPageImpl const* page, const common::Span<RegTree::Node> d_nodes,
      common::Span<FeatureType const> d_feature_types, common::Span<uint32_t const> categories,
      common::Span<RegTree::CategoricalSplitMatrix::Segment> categories_segments,
      HostDeviceVector<bst_node_t>* p_out_position) {
    auto d_matrix = page->GetDeviceAccessor(ctx_->gpu_id);
    auto d_gpair = this->gpair;
    p_out_position->SetDevice(ctx_->gpu_id);
    p_out_position->Resize(row_partitioner->GetRows().size());

    auto new_position_op = [=] __device__(size_t row_id, int position) {
      // What happens if user prune the tree?
      if (!d_matrix.IsInRange(row_id)) {
        return RowPartitioner::kIgnoredTreePosition;
      }
      auto node = d_nodes[position];

      while (!node.IsLeaf()) {
        bst_float element = d_matrix.GetFvalue(row_id, node.SplitIndex());
        // Missing value
        if (isnan(element)) {
          position = node.DefaultChild();
        } else {
          bool go_left = true;
          if (common::IsCat(d_feature_types, position)) {
            auto node_cats = categories.subspan(categories_segments[position].beg,
                                                categories_segments[position].size);
            go_left = common::Decision(node_cats, element);
          } else {
            go_left = element <= node.SplitCond();
          }
          if (go_left) {
            position = node.LeftChild();
          } else {
            position = node.RightChild();
          }
        }

        node = d_nodes[position];
      }

      return position;
    };  // NOLINT

    auto d_out_position = p_out_position->DeviceSpan();
    row_partitioner->FinalisePosition(d_out_position, new_position_op);

    auto s_position = p_out_position->ConstDeviceSpan();
    positions.resize(s_position.size());
    dh::safe_cuda(hipMemcpyAsync(positions.data().get(), s_position.data(),
                                  s_position.size_bytes(), hipMemcpyDeviceToDevice,
                                  ctx_->CUDACtx()->Stream()));

    dh::LaunchN(row_partitioner->GetRows().size(), [=] __device__(size_t idx) {
      bst_node_t position = d_out_position[idx];
      bool is_row_sampled = d_gpair[idx].GetHess() - .0f == 0.f;
      d_out_position[idx] = is_row_sampled ? ~position : position;
    });
  }

  bool UpdatePredictionCache(linalg::MatrixView<float> out_preds_d, RegTree const* p_tree) {
    if (positions.empty()) {
      return false;
    }

    CHECK(p_tree);
    dh::safe_cuda(hipSetDevice(ctx_->gpu_id));
    CHECK_EQ(out_preds_d.DeviceIdx(), ctx_->gpu_id);

    auto d_position = dh::ToSpan(positions);
    CHECK_EQ(out_preds_d.Size(), d_position.size());

    auto const& h_nodes = p_tree->GetNodes();
    dh::caching_device_vector<RegTree::Node> nodes(h_nodes.size());
    dh::safe_cuda(hipMemcpyAsync(nodes.data().get(), h_nodes.data(),
                                  h_nodes.size() * sizeof(RegTree::Node), hipMemcpyHostToDevice,
                                  ctx_->CUDACtx()->Stream()));
    auto d_nodes = dh::ToSpan(nodes);
    CHECK_EQ(out_preds_d.Shape(1), 1);
    dh::LaunchN(d_position.size(), ctx_->CUDACtx()->Stream(),
                [=] XGBOOST_DEVICE(std::size_t idx) mutable {
                  bst_node_t nidx = d_position[idx];
                  auto weight = d_nodes[nidx].LeafValue();
                  out_preds_d(idx, 0) += weight;
                });
    return true;
  }

  // num histograms is the number of contiguous histograms in memory to reduce over
  void AllReduceHist(int nidx, int num_histograms) {
    monitor.Start("AllReduce");
    auto d_node_hist = hist.GetNodeHistogram(nidx).data();
    using ReduceT = typename std::remove_pointer<decltype(d_node_hist)>::type::ValueT;
    collective::AllReduce<collective::Operation::kSum>(
        ctx_->gpu_id, reinterpret_cast<ReduceT*>(d_node_hist),
        page->Cuts().TotalBins() * 2 * num_histograms);

    monitor.Stop("AllReduce");
  }

  /**
   * \brief Build GPU local histograms for the left and right child of some parent node
   */
  void BuildHistLeftRight(std::vector<GPUExpandEntry> const& candidates, const RegTree& tree) {
    if (candidates.empty()) return;
    // Some nodes we will manually compute histograms
    // others we will do by subtraction
    std::vector<int> hist_nidx;
    std::vector<int> subtraction_nidx;
    for (auto& e : candidates) {
      // Decide whether to build the left histogram or right histogram
      // Use sum of Hessian as a heuristic to select node with fewest training instances
      bool fewer_right = e.split.right_sum.GetQuantisedHess() < e.split.left_sum.GetQuantisedHess();
      if (fewer_right) {
        hist_nidx.emplace_back(tree[e.nid].RightChild());
        subtraction_nidx.emplace_back(tree[e.nid].LeftChild());
      } else {
        hist_nidx.emplace_back(tree[e.nid].LeftChild());
        subtraction_nidx.emplace_back(tree[e.nid].RightChild());
      }
    }
    std::vector<int> all_new = hist_nidx;
    all_new.insert(all_new.end(), subtraction_nidx.begin(), subtraction_nidx.end());
    // Allocate the histograms
    // Guaranteed contiguous memory
    hist.AllocateHistograms(all_new);

    for (auto nidx : hist_nidx) {
      this->BuildHist(nidx);
    }

    // Reduce all in one go
    // This gives much better latency in a distributed setting
    // when processing a large batch
    this->AllReduceHist(hist_nidx.at(0), hist_nidx.size());

    for (size_t i = 0; i < subtraction_nidx.size(); i++) {
      auto build_hist_nidx = hist_nidx.at(i);
      auto subtraction_trick_nidx = subtraction_nidx.at(i);
      auto parent_nidx = candidates.at(i).nid;

      if (!this->SubtractionTrick(parent_nidx, build_hist_nidx, subtraction_trick_nidx)) {
        // Calculate other histogram manually
        this->BuildHist(subtraction_trick_nidx);
        this->AllReduceHist(subtraction_trick_nidx, 1);
      }
    }
  }

  void ApplySplit(const GPUExpandEntry& candidate, RegTree* p_tree) {
    RegTree& tree = *p_tree;

    // Sanity check - have we created a leaf with no training instances?
    if (!collective::IsDistributed() && row_partitioner) {
      CHECK(row_partitioner->GetRows(candidate.nid).size() > 0)
          << "No training instances in this leaf!";
    }

    auto base_weight = candidate.base_weight;
    auto left_weight = candidate.left_weight * param.learning_rate;
    auto right_weight = candidate.right_weight * param.learning_rate;
    auto parent_hess = quantiser
                           ->ToFloatingPoint(candidate.split.left_sum +
                                             candidate.split.right_sum)
                           .GetHess();
    auto left_hess =
        quantiser->ToFloatingPoint(candidate.split.left_sum).GetHess();
    auto right_hess =
        quantiser->ToFloatingPoint(candidate.split.right_sum).GetHess();

    auto is_cat = candidate.split.is_cat;
    if (is_cat) {
      // should be set to nan in evaluation split.
      CHECK(common::CheckNAN(candidate.split.fvalue));
      std::vector<common::CatBitField::value_type> split_cats;

      auto h_cats = this->evaluator_.GetHostNodeCats(candidate.nid);
      auto n_bins_feature = page->Cuts().FeatureBins(candidate.split.findex);
      split_cats.resize(common::CatBitField::ComputeStorageSize(n_bins_feature), 0);
      CHECK_LE(split_cats.size(), h_cats.size());
      std::copy(h_cats.data(), h_cats.data() + split_cats.size(), split_cats.data());

      tree.ExpandCategorical(
          candidate.nid, candidate.split.findex, split_cats, candidate.split.dir == kLeftDir,
          base_weight, left_weight, right_weight, candidate.split.loss_chg, parent_hess,
          left_hess, right_hess);
    } else {
      CHECK(!common::CheckNAN(candidate.split.fvalue));
      tree.ExpandNode(candidate.nid, candidate.split.findex, candidate.split.fvalue,
                      candidate.split.dir == kLeftDir, base_weight, left_weight, right_weight,
                      candidate.split.loss_chg, parent_hess,
          left_hess, right_hess);
    }
    evaluator_.ApplyTreeSplit(candidate, p_tree);

    const auto& parent = tree[candidate.nid];
    interaction_constraints.Split(candidate.nid, parent.SplitIndex(), parent.LeftChild(),
                                  parent.RightChild());
  }

  GPUExpandEntry InitRoot(RegTree* p_tree) {
    constexpr bst_node_t kRootNIdx = 0;
    dh::XGBCachingDeviceAllocator<char> alloc;
    auto quantiser = *this->quantiser;
    auto gpair_it = dh::MakeTransformIterator<GradientPairInt64>(
        dh::tbegin(gpair), [=] __device__(auto const &gpair) {
          return quantiser.ToFixedPoint(gpair);
        });
    GradientPairInt64 root_sum_quantised =
        dh::Reduce(ctx_->CUDACtx()->CTP(), gpair_it, gpair_it + gpair.size(),
                   GradientPairInt64{}, thrust::plus<GradientPairInt64>{});
    using ReduceT = typename decltype(root_sum_quantised)::ValueT;
    collective::Allreduce<collective::Operation::kSum>(
        reinterpret_cast<ReduceT *>(&root_sum_quantised), 2);

    hist.AllocateHistograms({kRootNIdx});
    this->BuildHist(kRootNIdx);
    this->AllReduceHist(kRootNIdx, 1);

    // Remember root stats
    auto root_sum = quantiser.ToFloatingPoint(root_sum_quantised);
    p_tree->Stat(kRootNIdx).sum_hess = root_sum.GetHess();
    auto weight = CalcWeight(param, root_sum);
    p_tree->Stat(kRootNIdx).base_weight = weight;
    (*p_tree)[kRootNIdx].SetLeaf(param.learning_rate * weight);

    // Generate first split
    auto root_entry = this->EvaluateRootSplit(root_sum_quantised);
    return root_entry;
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair_all, DMatrix* p_fmat, ObjInfo const* task,
                  RegTree* p_tree, HostDeviceVector<bst_node_t>* p_out_position) {
    auto& tree = *p_tree;
    // Process maximum 32 nodes at a time
    Driver<GPUExpandEntry> driver(param, 32);

    monitor.Start("Reset");
    this->Reset(gpair_all, p_fmat, p_fmat->Info().num_col_);
    monitor.Stop("Reset");

    monitor.Start("InitRoot");
    driver.Push({this->InitRoot(p_tree)});
    monitor.Stop("InitRoot");

    // The set of leaves that can be expanded asynchronously
    auto expand_set = driver.Pop();
    while (!expand_set.empty()) {
      for (auto& candidate : expand_set) {
        this->ApplySplit(candidate, p_tree);
      }
      // Get the candidates we are allowed to expand further
      // e.g. We do not bother further processing nodes whose children are beyond max depth
      std::vector<GPUExpandEntry> filtered_expand_set;
      std::copy_if(expand_set.begin(), expand_set.end(), std::back_inserter(filtered_expand_set),
                   [&](const auto& e) { return driver.IsChildValid(e); });

      auto new_candidates =
          pinned.GetSpan<GPUExpandEntry>(filtered_expand_set.size() * 2, GPUExpandEntry());

      monitor.Start("UpdatePosition");
      // Update position is only run when child is valid, instead of right after apply
      // split (as in approx tree method).  Hense we have the finalise position call
      // in GPU Hist.
      this->UpdatePosition(filtered_expand_set, p_tree);
      monitor.Stop("UpdatePosition");

      monitor.Start("BuildHist");
      this->BuildHistLeftRight(filtered_expand_set, tree);
      monitor.Stop("BuildHist");

      monitor.Start("EvaluateSplits");
      this->EvaluateSplits(filtered_expand_set, *p_tree, new_candidates);
      monitor.Stop("EvaluateSplits");
      dh::DefaultStream().Sync();
      driver.Push(new_candidates.begin(), new_candidates.end());
      expand_set = driver.Pop();
    }

    monitor.Start("FinalisePosition");
    this->FinalisePosition(p_tree, p_fmat, *task, p_out_position);
    monitor.Stop("FinalisePosition");
  }
};

class GPUHistMaker : public TreeUpdater {
  using GradientSumT = GradientPairPrecise;

 public:
  explicit GPUHistMaker(Context const* ctx, ObjInfo const* task) : TreeUpdater(ctx), task_{task} {};
  void Configure(const Args& args) override {
    // Used in test to count how many configurations are performed
    LOG(DEBUG) << "[GPU Hist]: Configure";
    hist_maker_param_.UpdateAllowUnknown(args);
    dh::CheckComputeCapability();
    initialised_ = false;

    monitor_.Init("updater_gpu_hist");
  }

  void LoadConfig(Json const& in) override {
    auto const& config = get<Object const>(in);
    FromJson(config.at("hist_train_param"), &this->hist_maker_param_);
    initialised_ = false;
  }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["hist_train_param"] = ToJson(hist_maker_param_);
  }

  ~GPUHistMaker() {  // NOLINT
    dh::GlobalMemoryLogger().Log();
  }

  void Update(TrainParam const* param, HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
              common::Span<HostDeviceVector<bst_node_t>> out_position,
              const std::vector<RegTree*>& trees) override {
    monitor_.Start("Update");

    // build tree
    try {
      std::size_t t_idx{0};
      for (xgboost::RegTree* tree : trees) {
        this->UpdateTree(param, gpair, dmat, tree, &out_position[t_idx]);
        this->hist_maker_param_.CheckTreesSynchronized(tree);
        ++t_idx;
      }
      dh::safe_cuda(hipGetLastError());
    } catch (const std::exception& e) {
      LOG(FATAL) << "Exception in gpu_hist: " << e.what() << std::endl;
    }
    monitor_.Stop("Update");
  }

  void InitDataOnce(TrainParam const* param, DMatrix* dmat) {
    CHECK_GE(ctx_->gpu_id, 0) << "Must have at least one device";
    info_ = &dmat->Info();

    // Synchronise the column sampling seed
    uint32_t column_sampling_seed = common::GlobalRandom()();
    collective::Broadcast(&column_sampling_seed, sizeof(column_sampling_seed), 0);
    this->column_sampler_ = std::make_shared<common::ColumnSampler>(column_sampling_seed);

    auto batch_param = BatchParam{param->max_bin, TrainParam::DftSparseThreshold()};
    dh::safe_cuda(hipSetDevice(ctx_->gpu_id));
    info_->feature_types.SetDevice(ctx_->gpu_id);
    maker = std::make_unique<GPUHistMakerDevice>(
        ctx_, !dmat->SingleColBlock(), info_->feature_types.ConstDeviceSpan(), info_->num_row_,
        *param, column_sampler_, info_->num_col_, batch_param);

    p_last_fmat_ = dmat;
    initialised_ = true;
  }

  void InitData(TrainParam const* param, DMatrix* dmat, RegTree const* p_tree) {
    if (!initialised_) {
      monitor_.Start("InitDataOnce");
      this->InitDataOnce(param, dmat);
      monitor_.Stop("InitDataOnce");
    }
    p_last_tree_ = p_tree;
    CHECK(hist_maker_param_.GetInitialised());
  }

  void UpdateTree(TrainParam const* param, HostDeviceVector<GradientPair>* gpair, DMatrix* p_fmat,
                  RegTree* p_tree, HostDeviceVector<bst_node_t>* p_out_position) {
    monitor_.Start("InitData");
    this->InitData(param, p_fmat, p_tree);
    monitor_.Stop("InitData");

    gpair->SetDevice(ctx_->gpu_id);
    maker->UpdateTree(gpair, p_fmat, task_, p_tree, p_out_position);
  }

  bool UpdatePredictionCache(const DMatrix* data,
                             linalg::MatrixView<bst_float> p_out_preds) override {
    if (maker == nullptr || p_last_fmat_ == nullptr || p_last_fmat_ != data) {
      return false;
    }
    monitor_.Start("UpdatePredictionCache");
    bool result = maker->UpdatePredictionCache(p_out_preds, p_last_tree_);
    monitor_.Stop("UpdatePredictionCache");
    return result;
  }

  MetaInfo* info_{};  // NOLINT

  std::unique_ptr<GPUHistMakerDevice> maker;  // NOLINT

  [[nodiscard]] char const* Name() const override { return "grow_gpu_hist"; }
  [[nodiscard]] bool HasNodePosition() const override { return true; }

 private:
  bool initialised_{false};

  HistMakerTrainParam hist_maker_param_;

  DMatrix* p_last_fmat_{nullptr};
  RegTree const* p_last_tree_{nullptr};
  ObjInfo const* task_{nullptr};

  common::Monitor monitor_;
  std::shared_ptr<common::ColumnSampler> column_sampler_;
};

#if !defined(GTEST_TEST)
XGBOOST_REGISTER_TREE_UPDATER(GPUHistMaker, "grow_gpu_hist")
    .describe("Grow tree with GPU.")
    .set_body([](Context const* ctx, ObjInfo const* task) {
      return new GPUHistMaker(ctx, task);
    });
#endif  // !defined(GTEST_TEST)

class GPUGlobalApproxMaker : public TreeUpdater {
 public:
  explicit GPUGlobalApproxMaker(Context const* ctx, ObjInfo const* task)
      : TreeUpdater(ctx), task_{task} {};
  void Configure(Args const& args) override {
    // Used in test to count how many configurations are performed
    LOG(DEBUG) << "[GPU Approx]: Configure";
    hist_maker_param_.UpdateAllowUnknown(args);
    if (hist_maker_param_.max_cached_hist_node != HistMakerTrainParam::DefaultNodes()) {
      LOG(WARNING) << "The `max_cached_hist_node` is ignored in GPU.";
    }
    dh::CheckComputeCapability();
    initialised_ = false;

    monitor_.Init(this->Name());
  }

  void LoadConfig(Json const& in) override {
    auto const& config = get<Object const>(in);
    FromJson(config.at("hist_train_param"), &this->hist_maker_param_);
    initialised_ = false;
  }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["hist_train_param"] = ToJson(hist_maker_param_);
  }
  ~GPUGlobalApproxMaker() override { dh::GlobalMemoryLogger().Log(); }

  void Update(TrainParam const* param, HostDeviceVector<GradientPair>* gpair, DMatrix* p_fmat,
              common::Span<HostDeviceVector<bst_node_t>> out_position,
              const std::vector<RegTree*>& trees) override {
    monitor_.Start("Update");

    this->InitDataOnce(p_fmat);
    // build tree
    hess_.resize(gpair->Size());
    auto hess = dh::ToSpan(hess_);

    gpair->SetDevice(ctx_->Device());
    auto d_gpair = gpair->ConstDeviceSpan();
    auto cuctx = ctx_->CUDACtx();
    thrust::transform(cuctx->CTP(), dh::tcbegin(d_gpair), dh::tcend(d_gpair), dh::tbegin(hess),
                      [=] XGBOOST_DEVICE(GradientPair const& g) { return g.GetHess(); });

    auto const& info = p_fmat->Info();
    info.feature_types.SetDevice(ctx_->Device());
    auto batch = BatchParam{param->max_bin, hess, !task_->const_hess};
    maker_ = std::make_unique<GPUHistMakerDevice>(
        ctx_, !p_fmat->SingleColBlock(), info.feature_types.ConstDeviceSpan(), info.num_row_,
        *param, column_sampler_, info.num_col_, batch);

    std::size_t t_idx{0};
    for (xgboost::RegTree* tree : trees) {
      this->UpdateTree(gpair, p_fmat, tree, &out_position[t_idx]);
      this->hist_maker_param_.CheckTreesSynchronized(tree);
      ++t_idx;
    }

    monitor_.Stop("Update");
  }

  void InitDataOnce(DMatrix* p_fmat) {
    if (this->initialised_) {
      return;
    }

    monitor_.Start(__func__);
    CHECK(ctx_->IsCUDA()) << error::InvalidCUDAOrdinal();
    // Synchronise the column sampling seed
    uint32_t column_sampling_seed = common::GlobalRandom()();
    collective::Broadcast(&column_sampling_seed, sizeof(column_sampling_seed), 0);
    this->column_sampler_ = std::make_shared<common::ColumnSampler>(column_sampling_seed);

    p_last_fmat_ = p_fmat;
    initialised_ = true;
    monitor_.Stop(__func__);
  }

  void InitData(DMatrix* p_fmat, RegTree const* p_tree) {
    this->InitDataOnce(p_fmat);
    p_last_tree_ = p_tree;
    CHECK(hist_maker_param_.GetInitialised());
  }

  void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* p_fmat, RegTree* p_tree,
                  HostDeviceVector<bst_node_t>* p_out_position) {
    monitor_.Start("InitData");
    this->InitData(p_fmat, p_tree);
    monitor_.Stop("InitData");

    gpair->SetDevice(ctx_->gpu_id);
    maker_->UpdateTree(gpair, p_fmat, task_, p_tree, p_out_position);
  }

  bool UpdatePredictionCache(const DMatrix* data,
                             linalg::MatrixView<bst_float> p_out_preds) override {
    if (maker_ == nullptr || p_last_fmat_ == nullptr || p_last_fmat_ != data) {
      return false;
    }
    monitor_.Start("UpdatePredictionCache");
    bool result = maker_->UpdatePredictionCache(p_out_preds, p_last_tree_);
    monitor_.Stop("UpdatePredictionCache");
    return result;
  }

  [[nodiscard]] char const* Name() const override { return "grow_gpu_approx"; }
  [[nodiscard]] bool HasNodePosition() const override { return true; }

 private:
  bool initialised_{false};

  HistMakerTrainParam hist_maker_param_;
  dh::device_vector<float> hess_;
  std::shared_ptr<common::ColumnSampler> column_sampler_;
  std::unique_ptr<GPUHistMakerDevice> maker_;

  DMatrix* p_last_fmat_{nullptr};
  RegTree const* p_last_tree_{nullptr};
  ObjInfo const* task_{nullptr};

  common::Monitor monitor_;
};

#if !defined(GTEST_TEST)
XGBOOST_REGISTER_TREE_UPDATER(GPUApproxMaker, "grow_gpu_approx")
    .describe("Grow tree with GPU.")
    .set_body([](Context const* ctx, ObjInfo const* task) {
      return new GPUGlobalApproxMaker(ctx, task);
    });
#endif  // !defined(GTEST_TEST)
}  // namespace xgboost::tree
