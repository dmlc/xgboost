#include "hip/hip_runtime.h"
/*!
 * Copyright 2020-2022 by XGBoost Contributors
 */
#include <algorithm>  // std::max
#include <vector>
#include <limits>

#include "../../common/categorical.h"
#include "../../common/device_helpers.cuh"
#include "../../data/ellpack_page.cuh"
#include "evaluate_splits.cuh"
#include "expand_entry.cuh"

namespace xgboost {
namespace tree {

// With constraints
XGBOOST_DEVICE float LossChangeMissing(const GradientPairPrecise &scan,
                                       const GradientPairPrecise &missing,
                                       const GradientPairPrecise &parent_sum,
                                       const GPUTrainingParam &param, bst_node_t nidx,
                                       bst_feature_t fidx,
                                       TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator,
                                       bool &missing_left_out) {  // NOLINT
  const auto left_sum = scan + missing;
  float missing_left_gain =
      evaluator.CalcSplitGain(param, nidx, fidx, left_sum, parent_sum - left_sum);
  float missing_right_gain = evaluator.CalcSplitGain(param, nidx, fidx, scan, parent_sum - scan);

  missing_left_out = missing_left_gain > missing_right_gain;
  return missing_left_out?missing_left_gain:missing_right_gain;
}

// This kernel uses block_size == warp_size. This is an unusually small block size for a cuda kernel
// - normally a larger block size is preferred to increase the number of resident warps on each SM
// (occupancy). In the below case each thread has a very large amount of work per thread relative to
// typical cuda kernels. Thus the SM can be highly utilised by a small number of threads. It was
// discovered by experiments that a small block size here is significantly faster. Furthermore,
// using only a single warp, synchronisation barriers are eliminated and broadcasts can be performed
// using warp intrinsics instead of slower shared memory.
template <int kBlockSize>
class EvaluateSplitAgent {
 public:
  using ArgMaxT = hipcub::KeyValuePair<int, float>;
  using BlockScanT = hipcub::BlockScan<GradientPairPrecise, kBlockSize>;
  using MaxReduceT = hipcub::WarpReduce<ArgMaxT>;
  using SumReduceT = hipcub::WarpReduce<GradientPairPrecise>;

  struct TempStorage {
    typename BlockScanT::TempStorage scan;
    typename MaxReduceT::TempStorage max_reduce;
    typename SumReduceT::TempStorage sum_reduce;
  };

  const int fidx;
  const int nidx;
  const float min_fvalue;
  const uint32_t gidx_begin;  // beginning bin
  const uint32_t gidx_end;    // end bin for i^th feature
  const dh::LDGIterator<float> feature_values;
  const GradientPairPrecise *node_histogram;
  const GradientPairPrecise parent_sum;
  const GradientPairPrecise missing;
  const GPUTrainingParam &param;
  const TreeEvaluator::SplitEvaluator<GPUTrainingParam> &evaluator;
  TempStorage *temp_storage;
  SumCallbackOp<GradientPairPrecise> prefix_op;
  static float constexpr kNullGain = -std::numeric_limits<bst_float>::infinity();

  __device__ EvaluateSplitAgent(TempStorage *temp_storage, int fidx,
                                const EvaluateSplitInputs &inputs,
                                const EvaluateSplitSharedInputs &shared_inputs,
                                const TreeEvaluator::SplitEvaluator<GPUTrainingParam> &evaluator)
      : temp_storage(temp_storage),
        nidx(inputs.nidx),
        fidx(fidx),
        min_fvalue(__ldg(shared_inputs.min_fvalue.data() + fidx)),
        gidx_begin(__ldg(shared_inputs.feature_segments.data() + fidx)),
        gidx_end(__ldg(shared_inputs.feature_segments.data() + fidx + 1)),
        feature_values(shared_inputs.feature_values.data()),
        node_histogram(inputs.gradient_histogram.data()),
        parent_sum(dh::LDGIterator<GradientPairPrecise>(&inputs.parent_sum)[0]),
        param(shared_inputs.param),
        evaluator(evaluator),
        missing(parent_sum - ReduceFeature()) {
    static_assert(kBlockSize == 32,
                  "This kernel relies on the assumption block_size == warp_size");
  }
  __device__ GradientPairPrecise ReduceFeature() {
    GradientPairPrecise local_sum;
    for (int idx = gidx_begin + threadIdx.x; idx < gidx_end; idx += kBlockSize) {
      local_sum += LoadGpair(node_histogram + idx);
    }
    local_sum = SumReduceT(temp_storage->sum_reduce).Sum(local_sum);
    // Broadcast result from thread 0
    return {__shfl_sync(0xffffffff, local_sum.GetGrad(), 0),
            __shfl_sync(0xffffffff, local_sum.GetHess(), 0)};
  }

  // Load using efficient 128 vector load instruction
  __device__ __forceinline__ GradientPairPrecise LoadGpair(const GradientPairPrecise *ptr) {
    static_assert(sizeof(GradientPairPrecise) == sizeof(float4),
                  "Vector type size does not match gradient pair size.");
    float4 tmp = *reinterpret_cast<const float4 *>(ptr);
    return *reinterpret_cast<const GradientPairPrecise *>(&tmp);
  }

  __device__ __forceinline__ void Numerical(DeviceSplitCandidate *__restrict__ best_split) {
    for (int scan_begin = gidx_begin; scan_begin < gidx_end; scan_begin += kBlockSize) {
      bool thread_active = (scan_begin + threadIdx.x) < gidx_end;
      GradientPairPrecise bin = thread_active ? LoadGpair(node_histogram + scan_begin + threadIdx.x)
                                              : GradientPairPrecise();
      BlockScanT(temp_storage->scan).ExclusiveScan(bin, bin, hipcub::Sum(), prefix_op);
      // Whether the gradient of missing values is put to the left side.
      bool missing_left = true;
      float gain = thread_active ? LossChangeMissing(bin, missing, parent_sum, param, nidx, fidx,
                                                     evaluator, missing_left)
                                 : kNullGain;

      // Find thread with best gain
      auto best = MaxReduceT(temp_storage->max_reduce).Reduce({threadIdx.x, gain}, hipcub::ArgMax());
      // This reduce result is only valid in thread 0
      // broadcast to the rest of the warp
      auto best_thread = __shfl_sync(0xffffffff, best.key, 0);

      // Best thread updates the split
      if (threadIdx.x == best_thread) {
        // Use pointer from cut to indicate begin and end of bins for each feature.
        int split_gidx = (scan_begin + threadIdx.x) - 1;
        float fvalue =
            split_gidx < static_cast<int>(gidx_begin) ? min_fvalue : feature_values[split_gidx];
        GradientPairPrecise left = missing_left ? bin + missing : bin;
        GradientPairPrecise right = parent_sum - left;
        best_split->Update(gain, missing_left ? kLeftDir : kRightDir, fvalue, fidx, left, right,
                           false, param);
      }
    }
  }

  __device__ __forceinline__ void OneHot(DeviceSplitCandidate *__restrict__ best_split) {
    for (int scan_begin = gidx_begin; scan_begin < gidx_end; scan_begin += kBlockSize) {
      bool thread_active = (scan_begin + threadIdx.x) < gidx_end;

      auto rest = thread_active ? LoadGpair(node_histogram + scan_begin + threadIdx.x)
                                : GradientPairPrecise();
      GradientPairPrecise bin = parent_sum - rest - missing;
      // Whether the gradient of missing values is put to the left side.
      bool missing_left = true;
      float gain = thread_active ? LossChangeMissing(bin, missing, parent_sum, param, nidx, fidx,
                                                     evaluator, missing_left)
                                 : kNullGain;

      // Find thread with best gain
      auto best = MaxReduceT(temp_storage->max_reduce).Reduce({threadIdx.x, gain}, hipcub::ArgMax());
      // This reduce result is only valid in thread 0
      // broadcast to the rest of the warp
      auto best_thread = __shfl_sync(0xffffffff, best.key, 0);
      // Best thread updates the split
      if (threadIdx.x == best_thread) {
        int32_t split_gidx = (scan_begin + threadIdx.x);
        float fvalue = feature_values[split_gidx];
        GradientPairPrecise left = missing_left ? bin + missing : bin;
        GradientPairPrecise right = parent_sum - left;
        best_split->UpdateCat(gain, missing_left ? kLeftDir : kRightDir,
                              static_cast<bst_cat_t>(fvalue), fidx, left, right, param);
      }
    }
  }
  /**
   * \brief Gather and update the best split.
   */
  __device__ __forceinline__ void PartitionUpdate(bst_bin_t scan_begin, bool thread_active,
                                                  bool missing_left, bst_bin_t it,
                                                  GradientPairPrecise const &left_sum,
                                                  GradientPairPrecise const &right_sum,
                                                  DeviceSplitCandidate *__restrict__ best_split) {
    auto gain =
        thread_active ? evaluator.CalcSplitGain(param, nidx, fidx, left_sum, right_sum) : kNullGain;

    // Find thread with best gain
    auto best = MaxReduceT(temp_storage->max_reduce).Reduce({threadIdx.x, gain}, hipcub::ArgMax());
    // This reduce result is only valid in thread 0
    // broadcast to the rest of the warp
    auto best_thread = __shfl_sync(0xffffffff, best.key, 0);
    // Best thread updates the split
    if (threadIdx.x == best_thread) {
      assert(thread_active);
      // index of best threshold inside a feature.
      auto best_thresh = it - gidx_begin;
      best_split->UpdateCat(gain, missing_left ? kLeftDir : kRightDir, best_thresh, fidx, left_sum,
                            right_sum, param);
    }
  }
  /**
   * \brief Partition-based split for categorical feature.
   */
  __device__ __forceinline__ void Partition(DeviceSplitCandidate *__restrict__ best_split,
                                            common::Span<bst_feature_t> sorted_idx,
                                            std::size_t node_offset,
                                            GPUTrainingParam const &param) {
    bst_bin_t n_bins_feature = gidx_end - gidx_begin;
    auto n_bins = std::min(param.max_cat_threshold, n_bins_feature);

    bst_bin_t it_begin = gidx_begin;
    bst_bin_t it_end = it_begin + n_bins - 1;

    // forward
    for (bst_bin_t scan_begin = it_begin; scan_begin < it_end; scan_begin += kBlockSize) {
      auto it = scan_begin + static_cast<bst_bin_t>(threadIdx.x);
      bool thread_active = it < it_end;

      auto right_sum = thread_active ? LoadGpair(node_histogram + sorted_idx[it] - node_offset)
                                     : GradientPairPrecise();
      // No min value for cat feature, use inclusive scan.
      BlockScanT(temp_storage->scan).InclusiveSum(right_sum, right_sum, prefix_op);
      GradientPairPrecise left_sum = parent_sum - right_sum;

      PartitionUpdate(scan_begin, thread_active, true, it, left_sum, right_sum, best_split);
    }

    // backward
    it_begin = gidx_end - 1;
    it_end = it_begin - n_bins + 1;
    prefix_op = SumCallbackOp<GradientPairPrecise>{};  // reset

    for (bst_bin_t scan_begin = it_begin; scan_begin > it_end; scan_begin -= kBlockSize) {
      auto it = scan_begin - static_cast<bst_bin_t>(threadIdx.x);
      bool thread_active = it > it_end;

      auto left_sum = thread_active ? LoadGpair(node_histogram + sorted_idx[it] - node_offset)
                                    : GradientPairPrecise();
      // No min value for cat feature, use inclusive scan.
      BlockScanT(temp_storage->scan).InclusiveSum(left_sum, left_sum, prefix_op);
      GradientPairPrecise right_sum = parent_sum - left_sum;

      PartitionUpdate(scan_begin, thread_active, false, it, left_sum, right_sum, best_split);
    }
  }
};

template <int kBlockSize>
__global__ __launch_bounds__(kBlockSize) void EvaluateSplitsKernel(
    bst_feature_t number_active_features, common::Span<const EvaluateSplitInputs> d_inputs,
    const EvaluateSplitSharedInputs shared_inputs, common::Span<bst_feature_t> sorted_idx,
    const TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator,
    common::Span<DeviceSplitCandidate> out_candidates) {
  // Aligned && shared storage for best_split
  __shared__ hipcub::Uninitialized<DeviceSplitCandidate> uninitialized_split;
  DeviceSplitCandidate &best_split = uninitialized_split.Alias();

  if (threadIdx.x == 0) {
    best_split = DeviceSplitCandidate();
  }

  __syncthreads();

  // Allocate blocks to one feature of one node
  const auto input_idx = blockIdx.x / number_active_features;
  const EvaluateSplitInputs &inputs = d_inputs[input_idx];
  // One block for each feature. Features are sampled, so fidx != blockIdx.x

  int fidx = inputs.feature_set[blockIdx.x % number_active_features];

  using AgentT = EvaluateSplitAgent<kBlockSize>;
  __shared__ typename AgentT::TempStorage temp_storage;
  AgentT agent(&temp_storage, fidx, inputs, shared_inputs, evaluator);

  if (common::IsCat(shared_inputs.feature_types, fidx)) {
    auto n_bins_in_feat =
        shared_inputs.feature_segments[fidx + 1] - shared_inputs.feature_segments[fidx];
    if (common::UseOneHot(n_bins_in_feat, shared_inputs.param.max_cat_to_onehot)) {
      agent.OneHot(&best_split);
    } else {
      auto total_bins = shared_inputs.feature_values.size();
      size_t offset = total_bins * input_idx;
      auto node_sorted_idx = sorted_idx.subspan(offset, total_bins);
      agent.Partition(&best_split, node_sorted_idx, offset, shared_inputs.param);
    }
  } else {
    agent.Numerical(&best_split);
  }

  hipcub::CTA_SYNC();
  if (threadIdx.x == 0) {
    // Record best loss for each feature
    out_candidates[blockIdx.x] = best_split;
  }
}

__device__ DeviceSplitCandidate operator+(const DeviceSplitCandidate &a,
                                          const DeviceSplitCandidate &b) {
  return b.loss_chg > a.loss_chg ? b : a;
}

/**
 * \brief Set the bits for categorical splits based on the split threshold.
 */
__device__ void SetCategoricalSplit(const EvaluateSplitSharedInputs &shared_inputs,
                                    common::Span<bst_feature_t const> d_sorted_idx,
                                    bst_feature_t fidx, std::size_t input_idx,
                                    common::Span<common::CatBitField::value_type> out,
                                    DeviceSplitCandidate *p_out_split) {
  auto &out_split = *p_out_split;
  out_split.split_cats = common::CatBitField{out};

  // Simple case for one hot split
  if (common::UseOneHot(shared_inputs.FeatureBins(fidx), shared_inputs.param.max_cat_to_onehot)) {
    out_split.split_cats.Set(common::AsCat(out_split.thresh));
    return;
  }

  // partition-based split
  auto node_sorted_idx = d_sorted_idx.subspan(shared_inputs.feature_values.size() * input_idx,
                                              shared_inputs.feature_values.size());
  size_t node_offset = input_idx * shared_inputs.feature_values.size();
  auto const best_thresh = out_split.thresh;
  if (best_thresh == -1) {
    return;
  }
  auto f_sorted_idx = node_sorted_idx.subspan(shared_inputs.feature_segments[fidx],
                                              shared_inputs.FeatureBins(fidx));
  bool forward = out_split.dir == kLeftDir;
  bst_bin_t partition = forward ? best_thresh + 1 : best_thresh;
  auto beg = dh::tcbegin(f_sorted_idx);
  assert(partition > 0 && "Invalid partition.");
  thrust::for_each(thrust::seq, beg, beg + partition, [&](size_t c) {
    auto cat = shared_inputs.feature_values[c - node_offset];
    out_split.SetCat(cat);
  });
}

void GPUHistEvaluator::LaunchEvaluateSplits(
    bst_feature_t number_active_features, common::Span<const EvaluateSplitInputs> d_inputs,
    EvaluateSplitSharedInputs shared_inputs,
    TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator,
    common::Span<DeviceSplitCandidate> out_splits) {
  if (need_sort_histogram_) {
    this->SortHistogram(d_inputs, shared_inputs, evaluator);
  }

  size_t combined_num_features = number_active_features * d_inputs.size();
  dh::TemporaryArray<DeviceSplitCandidate> feature_best_splits(combined_num_features);

  // One block for each feature
  uint32_t constexpr kBlockThreads = 32;
  dh::LaunchKernel {static_cast<uint32_t>(combined_num_features), kBlockThreads, 0}(
      EvaluateSplitsKernel<kBlockThreads>, number_active_features, d_inputs,
      shared_inputs, this->SortedIdx(d_inputs.size(), shared_inputs.feature_values.size()),
      evaluator, dh::ToSpan(feature_best_splits));

  // Reduce to get best candidate for left and right child over all features
  auto reduce_offset = dh::MakeTransformIterator<size_t>(
      thrust::make_counting_iterator(0llu),
      [=] __device__(size_t idx) -> size_t { return idx * number_active_features; });
  size_t temp_storage_bytes = 0;
  auto num_segments = out_splits.size();
  hipcub::DeviceSegmentedReduce::Sum(nullptr, temp_storage_bytes, feature_best_splits.data(),
                                  out_splits.data(), num_segments, reduce_offset,
                                  reduce_offset + 1);
  dh::TemporaryArray<int8_t> temp(temp_storage_bytes);
  hipcub::DeviceSegmentedReduce::Sum(temp.data().get(), temp_storage_bytes, feature_best_splits.data(),
                                  out_splits.data(), num_segments, reduce_offset,
                                  reduce_offset + 1);
}

void GPUHistEvaluator::CopyToHost(const std::vector<bst_node_t> &nidx) {
  if (!has_categoricals_) return;
  auto d_cats = this->DeviceCatStorage(nidx);
  auto h_cats = this->HostCatStorage(nidx);
  dh::CUDAEvent event;
  event.Record(dh::DefaultStream());
  for (auto idx : nidx) {
    copy_stream_.View().Wait(event);
    dh::safe_cuda(hipMemcpyAsync(
        h_cats.GetNodeCatStorage(idx).data(), d_cats.GetNodeCatStorage(idx).data(),
        d_cats.GetNodeCatStorage(idx).size_bytes(), hipMemcpyDeviceToHost, copy_stream_.View()));
  }
}

void GPUHistEvaluator::EvaluateSplits(
    const std::vector<bst_node_t> &nidx, bst_feature_t number_active_features,
    common::Span<const EvaluateSplitInputs> d_inputs, EvaluateSplitSharedInputs shared_inputs,
    common::Span<GPUExpandEntry> out_entries) {
  auto evaluator = this->tree_evaluator_.template GetEvaluator<GPUTrainingParam>();

  dh::TemporaryArray<DeviceSplitCandidate> splits_out_storage(d_inputs.size());
  auto out_splits = dh::ToSpan(splits_out_storage);
  this->LaunchEvaluateSplits(number_active_features, d_inputs, shared_inputs, evaluator,
                             out_splits);

  auto d_sorted_idx = this->SortedIdx(d_inputs.size(), shared_inputs.feature_values.size());
  auto d_entries = out_entries;
  auto device_cats_accessor = this->DeviceCatStorage(nidx);
  // turn candidate into entry, along with handling sort based split.
  dh::LaunchN(d_inputs.size(), [=] __device__(size_t i) mutable {
    auto const input = d_inputs[i];
    auto &split = out_splits[i];
    // Subtract parent gain here
    // As it is constant, this is more efficient than doing it during every split evaluation
    float parent_gain = CalcGain(shared_inputs.param, input.parent_sum);
    split.loss_chg -= parent_gain;
    auto fidx = out_splits[i].findex;

    if (split.is_cat) {
      SetCategoricalSplit(shared_inputs, d_sorted_idx, fidx, i,
                          device_cats_accessor.GetNodeCatStorage(input.nidx), &out_splits[i]);
    }

    float base_weight = evaluator.CalcWeight(input.nidx, shared_inputs.param,
                                             GradStats{split.left_sum + split.right_sum});
    float left_weight =
        evaluator.CalcWeight(input.nidx, shared_inputs.param, GradStats{split.left_sum});
    float right_weight =
        evaluator.CalcWeight(input.nidx, shared_inputs.param, GradStats{split.right_sum});

    d_entries[i] = GPUExpandEntry{input.nidx,  input.depth, out_splits[i],
                                  base_weight, left_weight, right_weight};
  });

  this->CopyToHost(nidx);
}

GPUExpandEntry GPUHistEvaluator::EvaluateSingleSplit(
    EvaluateSplitInputs input, EvaluateSplitSharedInputs shared_inputs) {
  dh::device_vector<EvaluateSplitInputs> inputs = std::vector<EvaluateSplitInputs>{input};
  dh::TemporaryArray<GPUExpandEntry> out_entries(1);
  this->EvaluateSplits({input.nidx}, input.feature_set.size(), dh::ToSpan(inputs), shared_inputs,
                       dh::ToSpan(out_entries));
  GPUExpandEntry root_entry;
  dh::safe_cuda(hipMemcpyAsync(&root_entry, out_entries.data().get(), sizeof(GPUExpandEntry),
                                hipMemcpyDeviceToHost));
  return root_entry;
}

}  // namespace tree
}  // namespace xgboost
