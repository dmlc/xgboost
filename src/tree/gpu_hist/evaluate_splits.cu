#include "hip/hip_runtime.h"
/*!
 * Copyright 2020-2022 by XGBoost Contributors
 */
#include <algorithm>  // std::max
#include <vector>
#include <limits>

#include "../../common/categorical.h"
#include "../../common/device_helpers.cuh"
#include "../../data/ellpack_page.cuh"
#include "evaluate_splits.cuh"
#include "expand_entry.cuh"

namespace xgboost {
namespace tree {

// With constraints
XGBOOST_DEVICE float LossChangeMissing(const GradientPairPrecise &scan,
                                       const GradientPairPrecise &missing,
                                       const GradientPairPrecise &parent_sum,
                                       const GPUTrainingParam &param, bst_node_t nidx,
                                       bst_feature_t fidx,
                                       TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator,
                                       bool &missing_left_out) {  // NOLINT
  float parent_gain = CalcGain(param, parent_sum);
  float missing_left_gain = evaluator.CalcSplitGain(param, nidx, fidx, GradStats(scan + missing),
                                                    GradStats(parent_sum - (scan + missing)));
  float missing_right_gain =
      evaluator.CalcSplitGain(param, nidx, fidx, GradStats(scan), GradStats(parent_sum - scan));

  if (missing_left_gain > missing_right_gain) {
    missing_left_out = true;
    return missing_left_gain - parent_gain;
  } else {
    missing_left_out = false;
    return missing_right_gain - parent_gain;
  }
}

/*!
 * \brief
 *
 * \tparam ReduceT     BlockReduce Type.
 * \tparam TempStorage Cub Shared memory
 *
 * \param begin
 * \param end
 * \param temp_storage Shared memory for intermediate result.
 */
template <int BLOCK_THREADS, typename ReduceT, typename TempStorageT, typename GradientSumT>
__device__ GradientSumT ReduceFeature(common::Span<const GradientSumT> feature_histogram,
                                      TempStorageT *temp_storage) {
  __shared__ hipcub::Uninitialized<GradientSumT> uninitialized_sum;
  GradientSumT &shared_sum = uninitialized_sum.Alias();

  GradientSumT local_sum = GradientSumT();
  // For loop sums features into one block size
  auto begin = feature_histogram.data();
  auto end = begin + feature_histogram.size();
  for (auto itr = begin; itr < end; itr += BLOCK_THREADS) {
    bool thread_active = itr + threadIdx.x < end;
    // Scan histogram
    GradientSumT bin = thread_active ? *(itr + threadIdx.x) : GradientSumT();
    local_sum += bin;
  }
  local_sum = ReduceT(temp_storage->sum_reduce).Reduce(local_sum, hipcub::Sum());
  // Reduction result is stored in thread 0.
  if (threadIdx.x == 0) {
    shared_sum = local_sum;
  }
  hipcub::CTA_SYNC();
  return shared_sum;
}

/*! \brief Find the thread with best gain. */
template <int BLOCK_THREADS, typename ReduceT, typename ScanT, typename MaxReduceT,
          typename TempStorageT, typename GradientSumT, SplitType type>
__device__ void EvaluateFeature(
    int fidx, const EvaluateSplitInputs &inputs, const EvaluateSplitSharedInputs &shared_inputs,
    TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator,
    common::Span<bst_feature_t> sorted_idx, size_t offset,
    DeviceSplitCandidate *best_split,  // shared memory storing best split
    TempStorageT *temp_storage         // temp memory for cub operations
) {
  // Use pointer from cut to indicate begin and end of bins for each feature.
  uint32_t gidx_begin = shared_inputs.feature_segments[fidx];    // beginning bin
  uint32_t gidx_end = shared_inputs.feature_segments[fidx + 1];  // end bin for i^th feature
  auto feature_hist = inputs.gradient_histogram.subspan(gidx_begin, gidx_end - gidx_begin);

  // Sum histogram bins for current feature
  GradientSumT const feature_sum =
      ReduceFeature<BLOCK_THREADS, ReduceT, TempStorageT, GradientSumT>(feature_hist, temp_storage);

  GradientPairPrecise const missing = inputs.parent_sum - GradientPairPrecise{feature_sum};
  float const null_gain = -std::numeric_limits<bst_float>::infinity();

  SumCallbackOp<GradientSumT> prefix_op = SumCallbackOp<GradientSumT>();
  for (int scan_begin = gidx_begin; scan_begin < gidx_end; scan_begin += BLOCK_THREADS) {
    bool thread_active = (scan_begin + threadIdx.x) < gidx_end;

    auto calc_bin_value = [&]() {
      GradientSumT bin;
      switch (type) {
        case kOneHot: {
          auto rest =
              thread_active ? inputs.gradient_histogram[scan_begin + threadIdx.x] : GradientSumT();
          bin = GradientSumT{inputs.parent_sum - GradientPairPrecise{rest} - missing};  // NOLINT
          break;
        }
        case kNum: {
          bin =
              thread_active ? inputs.gradient_histogram[scan_begin + threadIdx.x] : GradientSumT();
          ScanT(temp_storage->scan).ExclusiveScan(bin, bin, hipcub::Sum(), prefix_op);
          break;
        }
        case kPart: {
          auto rest = thread_active
                          ? inputs.gradient_histogram[sorted_idx[scan_begin + threadIdx.x] - offset]
                          : GradientSumT();
          // No min value for cat feature, use inclusive scan.
          ScanT(temp_storage->scan).InclusiveScan(rest, rest, hipcub::Sum(), prefix_op);
          bin = GradientSumT{inputs.parent_sum - GradientPairPrecise{rest} - missing};  // NOLINT
          break;
        }
      }
      return bin;
    };
    auto bin = calc_bin_value();
    // Whether the gradient of missing values is put to the left side.
    bool missing_left = true;
    float gain = null_gain;
    if (thread_active) {
      gain = LossChangeMissing(GradientPairPrecise{bin}, missing, inputs.parent_sum,
                               shared_inputs.param, inputs.nidx, fidx, evaluator, missing_left);
    }

    __syncthreads();

    // Find thread with best gain
    hipcub::KeyValuePair<int, float> tuple(threadIdx.x, gain);
    hipcub::KeyValuePair<int, float> best =
        MaxReduceT(temp_storage->max_reduce).Reduce(tuple, hipcub::ArgMax());

    __shared__ hipcub::KeyValuePair<int, float> block_max;
    if (threadIdx.x == 0) {
      block_max = best;
    }

    hipcub::CTA_SYNC();

    // Best thread updates the split
    if (threadIdx.x == block_max.key) {
      switch (type) {
        case kNum: {
          // Use pointer from cut to indicate begin and end of bins for each feature.
          uint32_t gidx_begin = shared_inputs.feature_segments[fidx];  // beginning bin
          int split_gidx = (scan_begin + threadIdx.x) - 1;
          float fvalue;
          if (split_gidx < static_cast<int>(gidx_begin)) {
            fvalue = shared_inputs.min_fvalue[fidx];
          } else {
            fvalue = shared_inputs.feature_values[split_gidx];
          }
          GradientPairPrecise left =
              missing_left ? GradientPairPrecise{bin} + missing : GradientPairPrecise{bin};
          GradientPairPrecise right = inputs.parent_sum - left;
          best_split->Update(gain, missing_left ? kLeftDir : kRightDir, fvalue, fidx, left, right,
                             false, shared_inputs.param);
          break;
        }
        case kOneHot: {
          int32_t split_gidx = (scan_begin + threadIdx.x);
          float fvalue = shared_inputs.feature_values[split_gidx];
          GradientPairPrecise left =
              missing_left ? GradientPairPrecise{bin} + missing : GradientPairPrecise{bin};
          GradientPairPrecise right = inputs.parent_sum - left;
          best_split->Update(gain, missing_left ? kLeftDir : kRightDir, fvalue, fidx, left, right,
                             true, shared_inputs.param);
          break;
        }
        case kPart: {
          int32_t split_gidx = (scan_begin + threadIdx.x);
          float fvalue = shared_inputs.feature_values[split_gidx];
          GradientPairPrecise left =
              missing_left ? GradientPairPrecise{bin} + missing : GradientPairPrecise{bin};
          GradientPairPrecise right = inputs.parent_sum - left;
          auto best_thresh = block_max.key;  // index of best threshold inside a feature.
          best_split->Update(gain, missing_left ? kLeftDir : kRightDir, best_thresh, fidx, left,
                             right, true, shared_inputs.param);
          break;
        }
      }
    }
    hipcub::CTA_SYNC();
  }
}

template <int BLOCK_THREADS, typename GradientSumT>
__global__ __launch_bounds__(BLOCK_THREADS) void EvaluateSplitsKernel(
    bst_feature_t number_active_features, common::Span<const EvaluateSplitInputs> d_inputs,
    const EvaluateSplitSharedInputs shared_inputs, common::Span<bst_feature_t> sorted_idx,
    TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator,
    common::Span<DeviceSplitCandidate> out_candidates) {
  // KeyValuePair here used as threadIdx.x -> gain_value
  using ArgMaxT = hipcub::KeyValuePair<int, float>;
  using BlockScanT = hipcub::BlockScan<GradientSumT, BLOCK_THREADS, hipcub::BLOCK_SCAN_WARP_SCANS>;
  using MaxReduceT = hipcub::BlockReduce<ArgMaxT, BLOCK_THREADS>;

  using SumReduceT = hipcub::BlockReduce<GradientSumT, BLOCK_THREADS>;

  union TempStorage {
    typename BlockScanT::TempStorage scan;
    typename MaxReduceT::TempStorage max_reduce;
    typename SumReduceT::TempStorage sum_reduce;
  };

  // Aligned && shared storage for best_split
  __shared__ hipcub::Uninitialized<DeviceSplitCandidate> uninitialized_split;
  DeviceSplitCandidate &best_split = uninitialized_split.Alias();
  __shared__ TempStorage temp_storage;

  if (threadIdx.x == 0) {
    best_split = DeviceSplitCandidate();
  }

  __syncthreads();

  // Allocate blocks to one feature of one node
  const auto input_idx = blockIdx.x / number_active_features;
  const EvaluateSplitInputs &inputs = d_inputs[input_idx];
  // One block for each feature. Features are sampled, so fidx != blockIdx.x

  int fidx = inputs.feature_set[blockIdx.x % number_active_features];

  if (common::IsCat(shared_inputs.feature_types, fidx)) {
    auto n_bins_in_feat =
        shared_inputs.feature_segments[fidx + 1] - shared_inputs.feature_segments[fidx];
    if (common::UseOneHot(n_bins_in_feat, shared_inputs.param.max_cat_to_onehot)) {
      EvaluateFeature<BLOCK_THREADS, SumReduceT, BlockScanT, MaxReduceT, TempStorage, GradientSumT,
                      kOneHot>(fidx, inputs, shared_inputs, evaluator, sorted_idx, 0, &best_split,
                               &temp_storage);
    } else {
      auto total_bins = shared_inputs.feature_values.size();
      size_t offset = total_bins * input_idx;
      auto node_sorted_idx = sorted_idx.subspan(offset, total_bins);
      EvaluateFeature<BLOCK_THREADS, SumReduceT, BlockScanT, MaxReduceT, TempStorage, GradientSumT,
                      kPart>(fidx, inputs, shared_inputs, evaluator, node_sorted_idx, offset,
                             &best_split, &temp_storage);
    }
  } else {
    EvaluateFeature<BLOCK_THREADS, SumReduceT, BlockScanT, MaxReduceT, TempStorage, GradientSumT,
                    kNum>(fidx, inputs, shared_inputs, evaluator, sorted_idx, 0, &best_split,
                          &temp_storage);
  }

  hipcub::CTA_SYNC();
  if (threadIdx.x == 0) {
    // Record best loss for each feature
    out_candidates[blockIdx.x] = best_split;
  }
}

__device__ DeviceSplitCandidate operator+(const DeviceSplitCandidate &a,
                                          const DeviceSplitCandidate &b) {
  return b.loss_chg > a.loss_chg ? b : a;
}

/**
 * \brief Set the bits for categorical splits based on the split threshold.
 */
__device__ void SetCategoricalSplit(const EvaluateSplitSharedInputs &shared_inputs,
                                    common::Span<bst_feature_t const> d_sorted_idx,
                                    bst_feature_t fidx, std::size_t input_idx,
                                    common::Span<common::CatBitField::value_type> out,
                                    DeviceSplitCandidate *p_out_split) {
  auto &out_split = *p_out_split;
  out_split.split_cats = common::CatBitField{out};

  // Simple case for one hot split
  if (common::UseOneHot(shared_inputs.FeatureBins(fidx), shared_inputs.param.max_cat_to_onehot)) {
    out_split.split_cats.Set(common::AsCat(out_split.fvalue));
    return;
  }

  auto node_sorted_idx = d_sorted_idx.subspan(shared_inputs.feature_values.size() * input_idx,
                                              shared_inputs.feature_values.size());
  size_t node_offset = input_idx * shared_inputs.feature_values.size();
  auto best_thresh = out_split.PopBestThresh();
  auto f_sorted_idx = node_sorted_idx.subspan(shared_inputs.feature_segments[fidx],
                                              shared_inputs.FeatureBins(fidx));
  if (out_split.dir != kLeftDir) {
    // forward, missing on right
    auto beg = dh::tcbegin(f_sorted_idx);
    // Don't put all the categories into one side
    auto boundary = std::min(static_cast<size_t>((best_thresh + 1)), (f_sorted_idx.size() - 1));
    boundary = std::max(boundary, static_cast<size_t>(1ul));
    auto end = beg + boundary;
    thrust::for_each(thrust::seq, beg, end, [&](auto c) {
      auto cat = shared_inputs.feature_values[c - node_offset];
      assert(!out_split.split_cats.Check(cat) && "already set");
      out_split.SetCat(cat);
    });
  } else {
    assert((f_sorted_idx.size() - best_thresh + 1) != 0 && " == 0");
    thrust::for_each(thrust::seq, dh::tcrbegin(f_sorted_idx),
                     dh::tcrbegin(f_sorted_idx) + (f_sorted_idx.size() - best_thresh), [&](auto c) {
                       auto cat = shared_inputs.feature_values[c - node_offset];
                       out_split.SetCat(cat);
                     });
  }
}

template <typename GradientSumT>
void GPUHistEvaluator<GradientSumT>::LaunchEvaluateSplits(
    bst_feature_t number_active_features, common::Span<const EvaluateSplitInputs> d_inputs,
    EvaluateSplitSharedInputs shared_inputs,
    TreeEvaluator::SplitEvaluator<GPUTrainingParam> evaluator,
    common::Span<DeviceSplitCandidate> out_splits) {
  if (need_sort_histogram_) {
    this->SortHistogram(d_inputs, shared_inputs, evaluator);
  }

  size_t combined_num_features = number_active_features * d_inputs.size();
  dh::TemporaryArray<DeviceSplitCandidate> feature_best_splits(combined_num_features);

  // One block for each feature
  uint32_t constexpr kBlockThreads = 32;
  dh::LaunchKernel {static_cast<uint32_t>(combined_num_features), kBlockThreads, 0}(
      EvaluateSplitsKernel<kBlockThreads, GradientSumT>, number_active_features, d_inputs,
      shared_inputs, this->SortedIdx(d_inputs.size(), shared_inputs.feature_values.size()),
      evaluator, dh::ToSpan(feature_best_splits));

  // Reduce to get best candidate for left and right child over all features
  auto reduce_offset = dh::MakeTransformIterator<size_t>(
      thrust::make_counting_iterator(0llu),
      [=] __device__(size_t idx) -> size_t { return idx * number_active_features; });
  size_t temp_storage_bytes = 0;
  auto num_segments = out_splits.size();
  hipcub::DeviceSegmentedReduce::Sum(nullptr, temp_storage_bytes, feature_best_splits.data(),
                                  out_splits.data(), num_segments, reduce_offset,
                                  reduce_offset + 1);
  dh::TemporaryArray<int8_t> temp(temp_storage_bytes);
  hipcub::DeviceSegmentedReduce::Sum(temp.data().get(), temp_storage_bytes, feature_best_splits.data(),
                                  out_splits.data(), num_segments, reduce_offset,
                                  reduce_offset + 1);
}

template <typename GradientSumT>
void GPUHistEvaluator<GradientSumT>::CopyToHost(const std::vector<bst_node_t> &nidx) {
  if (!has_categoricals_) return;
  auto d_cats = this->DeviceCatStorage(nidx);
  auto h_cats = this->HostCatStorage(nidx);
  dh::CUDAEvent event;
  event.Record(dh::DefaultStream());
  for (auto idx : nidx) {
    copy_stream_.View().Wait(event);
    dh::safe_cuda(hipMemcpyAsync(
        h_cats.GetNodeCatStorage(idx).data(), d_cats.GetNodeCatStorage(idx).data(),
        d_cats.GetNodeCatStorage(idx).size_bytes(), hipMemcpyDeviceToHost, copy_stream_.View()));
  }
}

template <typename GradientSumT>
void GPUHistEvaluator<GradientSumT>::EvaluateSplits(
    const std::vector<bst_node_t> &nidx, bst_feature_t number_active_features,
    common::Span<const EvaluateSplitInputs> d_inputs, EvaluateSplitSharedInputs shared_inputs,
    common::Span<GPUExpandEntry> out_entries) {
  auto evaluator = this->tree_evaluator_.template GetEvaluator<GPUTrainingParam>();

  dh::TemporaryArray<DeviceSplitCandidate> splits_out_storage(d_inputs.size());
  auto out_splits = dh::ToSpan(splits_out_storage);
  this->LaunchEvaluateSplits(number_active_features, d_inputs, shared_inputs, evaluator,
                             out_splits);

  auto d_sorted_idx = this->SortedIdx(d_inputs.size(), shared_inputs.feature_values.size());
  auto d_entries = out_entries;
  auto device_cats_accessor = this->DeviceCatStorage(nidx);
  // turn candidate into entry, along with handling sort based split.
  dh::LaunchN(d_inputs.size(), [=] __device__(size_t i) mutable {
    auto const input = d_inputs[i];
    auto &split = out_splits[i];
    auto fidx = out_splits[i].findex;

    if (split.is_cat) {
      SetCategoricalSplit(shared_inputs, d_sorted_idx, fidx, i,
                          device_cats_accessor.GetNodeCatStorage(input.nidx), &out_splits[i]);
    }

    float base_weight = evaluator.CalcWeight(input.nidx, shared_inputs.param,
                                             GradStats{split.left_sum + split.right_sum});
    float left_weight =
        evaluator.CalcWeight(input.nidx, shared_inputs.param, GradStats{split.left_sum});
    float right_weight =
        evaluator.CalcWeight(input.nidx, shared_inputs.param, GradStats{split.right_sum});

    d_entries[i] = GPUExpandEntry{input.nidx,  input.depth, out_splits[i],
                                  base_weight, left_weight, right_weight};
  });

  this->CopyToHost(nidx);
}

template <typename GradientSumT>
GPUExpandEntry GPUHistEvaluator<GradientSumT>::EvaluateSingleSplit(
    EvaluateSplitInputs input, EvaluateSplitSharedInputs shared_inputs) {
  dh::device_vector<EvaluateSplitInputs> inputs = std::vector<EvaluateSplitInputs>{input};
  dh::TemporaryArray<GPUExpandEntry> out_entries(1);
  this->EvaluateSplits({input.nidx}, input.feature_set.size(), dh::ToSpan(inputs), shared_inputs,
                       dh::ToSpan(out_entries));
  GPUExpandEntry root_entry;
  dh::safe_cuda(hipMemcpyAsync(&root_entry, out_entries.data().get(), sizeof(GPUExpandEntry),
                                hipMemcpyDeviceToHost));
  return root_entry;
}

template class GPUHistEvaluator<GradientPairPrecise>;
}  // namespace tree
}  // namespace xgboost
