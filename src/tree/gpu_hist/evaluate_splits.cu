#include "hip/hip_runtime.h"
/*!
 * Copyright 2020 by XGBoost Contributors
 */
#include "evaluate_splits.cuh"
#include <limits>

namespace xgboost {
namespace tree {

// With constraints
template <typename GradientPairT>
XGBOOST_DEVICE float LossChangeMissing(const GradientPairT& scan,
                                       const GradientPairT& missing,
                                       const GradientPairT& parent_sum,
                                       const GPUTrainingParam& param,
                                       int constraint,
                                       const ValueConstraint& value_constraint,
                                       bool& missing_left_out) {  // NOLINT
  float parent_gain = CalcGain(param, parent_sum);
  float missing_left_gain = value_constraint.CalcSplitGain(
      param, constraint, GradStats(scan + missing),
      GradStats(parent_sum - (scan + missing)));
  float missing_right_gain = value_constraint.CalcSplitGain(
      param, constraint, GradStats(scan), GradStats(parent_sum - scan));

  if (missing_left_gain >= missing_right_gain) {
    missing_left_out = true;
    return missing_left_gain - parent_gain;
  } else {
    missing_left_out = false;
    return missing_right_gain - parent_gain;
  }
}

/*!
 * \brief
 *
 * \tparam ReduceT     BlockReduce Type.
 * \tparam TempStorage Cub Shared memory
 *
 * \param begin
 * \param end
 * \param temp_storage Shared memory for intermediate result.
 */
template <int BLOCK_THREADS, typename ReduceT, typename TempStorageT,
          typename GradientSumT>
__device__ GradientSumT
ReduceFeature(common::Span<const GradientSumT> feature_histogram,
              TempStorageT* temp_storage) {
  __shared__ hipcub::Uninitialized<GradientSumT> uninitialized_sum;
  GradientSumT& shared_sum = uninitialized_sum.Alias();

  GradientSumT local_sum = GradientSumT();
  // For loop sums features into one block size
  auto begin = feature_histogram.data();
  auto end = begin + feature_histogram.size();
  for (auto itr = begin; itr < end; itr += BLOCK_THREADS) {
    bool thread_active = itr + threadIdx.x < end;
    // Scan histogram
    GradientSumT bin = thread_active ? *(itr + threadIdx.x) : GradientSumT();
    local_sum += bin;
  }
  local_sum = ReduceT(temp_storage->sum_reduce).Reduce(local_sum, hipcub::Sum());
  // Reduction result is stored in thread 0.
  if (threadIdx.x == 0) {
    shared_sum = local_sum;
  }
  __syncthreads();
  return shared_sum;
}

/*! \brief Find the thread with best gain. */
template <int BLOCK_THREADS, typename ReduceT, typename ScanT,
          typename MaxReduceT, typename TempStorageT, typename GradientSumT>
__device__ void EvaluateFeature(
    int fidx, EvaluateSplitInputs<GradientSumT> inputs,
    DeviceSplitCandidate* best_split,  // shared memory storing best split
    TempStorageT* temp_storage         // temp memory for cub operations
) {
  // Use pointer from cut to indicate begin and end of bins for each feature.
  uint32_t gidx_begin = inputs.feature_segments[fidx];  // begining bin
  uint32_t gidx_end =
      inputs.feature_segments[fidx + 1];  // end bin for i^th feature

  // Sum histogram bins for current feature
  GradientSumT const feature_sum =
      ReduceFeature<BLOCK_THREADS, ReduceT, TempStorageT, GradientSumT>(
          inputs.gradient_histogram.subspan(gidx_begin, gidx_end - gidx_begin),
          temp_storage);

  GradientSumT const missing = inputs.parent_sum - feature_sum;
  float const null_gain = -std::numeric_limits<bst_float>::infinity();

  SumCallbackOp<GradientSumT> prefix_op = SumCallbackOp<GradientSumT>();
  for (int scan_begin = gidx_begin; scan_begin < gidx_end;
       scan_begin += BLOCK_THREADS) {
    bool thread_active = (scan_begin + threadIdx.x) < gidx_end;

    // Gradient value for current bin.
    GradientSumT bin = thread_active
                           ? inputs.gradient_histogram[scan_begin + threadIdx.x]
                           : GradientSumT();
    ScanT(temp_storage->scan).ExclusiveScan(bin, bin, hipcub::Sum(), prefix_op);

    // Whether the gradient of missing values is put to the left side.
    bool missing_left = true;
    float gain = null_gain;
    if (thread_active) {
      gain = LossChangeMissing(bin, missing, inputs.parent_sum, inputs.param,
                               inputs.monotonic_constraints[fidx],
                               inputs.value_constraint, missing_left);
    }

    __syncthreads();

    // Find thread with best gain
    hipcub::KeyValuePair<int, float> tuple(threadIdx.x, gain);
    hipcub::KeyValuePair<int, float> best =
        MaxReduceT(temp_storage->max_reduce).Reduce(tuple, hipcub::ArgMax());

    __shared__ hipcub::KeyValuePair<int, float> block_max;
    if (threadIdx.x == 0) {
      block_max = best;
    }

    __syncthreads();

    // Best thread updates split
    if (threadIdx.x == block_max.key) {
      int split_gidx = (scan_begin + threadIdx.x) - 1;
      float fvalue;
      if (split_gidx < static_cast<int>(gidx_begin)) {
        fvalue = inputs.min_fvalue[fidx];
      } else {
        fvalue = inputs.feature_values[split_gidx];
      }
      GradientSumT left = missing_left ? bin + missing : bin;
      GradientSumT right = inputs.parent_sum - left;
      best_split->Update(gain, missing_left ? kLeftDir : kRightDir, fvalue,
                         fidx, GradientPair(left), GradientPair(right),
                         inputs.param);
    }
    __syncthreads();
  }
}

template <int BLOCK_THREADS, typename GradientSumT>
__global__ void EvaluateSplitsKernel(
    EvaluateSplitInputs<GradientSumT> left,
    EvaluateSplitInputs<GradientSumT> right,
    common::Span<DeviceSplitCandidate> out_candidates) {
  // KeyValuePair here used as threadIdx.x -> gain_value
  using ArgMaxT = hipcub::KeyValuePair<int, float>;
  using BlockScanT =
      hipcub::BlockScan<GradientSumT, BLOCK_THREADS, hipcub::BLOCK_SCAN_WARP_SCANS>;
  using MaxReduceT = hipcub::BlockReduce<ArgMaxT, BLOCK_THREADS>;

  using SumReduceT = hipcub::BlockReduce<GradientSumT, BLOCK_THREADS>;

  union TempStorage {
    typename BlockScanT::TempStorage scan;
    typename MaxReduceT::TempStorage max_reduce;
    typename SumReduceT::TempStorage sum_reduce;
  };

  // Aligned && shared storage for best_split
  __shared__ hipcub::Uninitialized<DeviceSplitCandidate> uninitialized_split;
  DeviceSplitCandidate& best_split = uninitialized_split.Alias();
  __shared__ TempStorage temp_storage;

  if (threadIdx.x == 0) {
    best_split = DeviceSplitCandidate();
  }

  __syncthreads();

  // If this block is working on the left or right node
  bool is_left = blockIdx.x < left.feature_set.size();
  EvaluateSplitInputs<GradientSumT>& inputs = is_left ? left : right;

  // One block for each feature. Features are sampled, so fidx != blockIdx.x
  int fidx = inputs.feature_set[is_left ? blockIdx.x
                                        : blockIdx.x - left.feature_set.size()];

  EvaluateFeature<BLOCK_THREADS, SumReduceT, BlockScanT, MaxReduceT>(
      fidx, inputs, &best_split, &temp_storage);

  __syncthreads();

  if (threadIdx.x == 0) {
    // Record best loss for each feature
    out_candidates[blockIdx.x] = best_split;
  }
}

__device__ DeviceSplitCandidate operator+(const DeviceSplitCandidate& a,
                                          const DeviceSplitCandidate& b) {
  return b.loss_chg > a.loss_chg ? b : a;
}

template <typename GradientSumT>
void EvaluateSplits(common::Span<DeviceSplitCandidate> out_splits,
                    EvaluateSplitInputs<GradientSumT> left,
                    EvaluateSplitInputs<GradientSumT> right) {
  size_t combined_num_features =
      left.feature_set.size() + right.feature_set.size();
  dh::TemporaryArray<DeviceSplitCandidate> feature_best_splits(
      combined_num_features);
  // One block for each feature
  uint32_t constexpr kBlockThreads = 256;
  dh::LaunchKernel {uint32_t(combined_num_features), kBlockThreads, 0}(
      EvaluateSplitsKernel<kBlockThreads, GradientSumT>, left, right,
      dh::ToSpan(feature_best_splits));

  // Reduce to get best candidate for left and right child over all features
  auto reduce_offset =
      dh::MakeTransformIterator<size_t>(thrust::make_counting_iterator(0llu),
                                        [=] __device__(size_t idx) -> size_t {
                                          if (idx == 0) {
                                            return 0;
                                          }
                                          if (idx == 1) {
                                            return left.feature_set.size();
                                          }
                                          if (idx == 2) {
                                            return combined_num_features;
                                          }
                                          return 0;
                                        });
  size_t temp_storage_bytes = 0;
  auto num_segments = out_splits.size();
  hipcub::DeviceSegmentedReduce::Sum(nullptr, temp_storage_bytes,
                                  feature_best_splits.data(), out_splits.data(),
                                  num_segments, reduce_offset, reduce_offset + 1);
  dh::TemporaryArray<int8_t> temp(temp_storage_bytes);
  hipcub::DeviceSegmentedReduce::Sum(temp.data().get(), temp_storage_bytes,
                                  feature_best_splits.data(), out_splits.data(),
                                  num_segments, reduce_offset, reduce_offset + 1);
}

template <typename GradientSumT>
void EvaluateSingleSplit(common::Span<DeviceSplitCandidate> out_split,
                         EvaluateSplitInputs<GradientSumT> input) {
  EvaluateSplits(out_split, input, {});
}

template void EvaluateSplits<GradientPair>(
    common::Span<DeviceSplitCandidate> out_splits,
    EvaluateSplitInputs<GradientPair> left,
    EvaluateSplitInputs<GradientPair> right);
template void EvaluateSplits<GradientPairPrecise>(
    common::Span<DeviceSplitCandidate> out_splits,
    EvaluateSplitInputs<GradientPairPrecise> left,
    EvaluateSplitInputs<GradientPairPrecise> right);
template void EvaluateSingleSplit<GradientPair>(
    common::Span<DeviceSplitCandidate> out_split,
    EvaluateSplitInputs<GradientPair> input);
template void EvaluateSingleSplit<GradientPairPrecise>(
    common::Span<DeviceSplitCandidate> out_split,
    EvaluateSplitInputs<GradientPairPrecise> input);
}  // namespace tree
}  // namespace xgboost
