#include "hip/hip_runtime.h"
/*!
 * Copyright 2019 by XGBoost Contributors
 */
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <xgboost/host_device_vector.h>
#include <xgboost/logging.h>

#include <algorithm>

#include "../../common/compressed_iterator.h"
#include "../../common/random.h"
#include "gradient_based_sampler.cuh"

namespace xgboost {
namespace tree {

GradientBasedSampler::GradientBasedSampler(EllpackPageImpl* page,
                                           size_t n_rows,
                                           BatchParam batch_param,
                                           float subsample,
                                           int sampling_method)
    : original_page_(page),
      batch_param_(batch_param),
      is_external_memory_(page->matrix.n_rows != n_rows),
      subsample_(subsample),
      is_sampling_(subsample < 1.0),
      sampling_method_(sampling_method),
      sample_rows_(n_rows * subsample) {
  monitor_.Init("gradient_based_sampler");

  if (is_sampling_ || is_external_memory_) {
    // Create a new ELLPACK page with empty rows.
    page_.reset(new EllpackPageImpl(batch_param.gpu_id,
                                    original_page_->matrix.info,
                                    sample_rows_));
  }
  // Allocate GPU memory for sampling.
  if (is_sampling_) {
    ba_.Allocate(batch_param_.gpu_id,
                 &gpair_, sample_rows_,
                 &row_weight_, n_rows,
                 &row_index_, n_rows,
                 &sample_row_index_, n_rows);
    thrust::copy(thrust::counting_iterator<size_t>(0),
                 thrust::counting_iterator<size_t>(n_rows),
                 dh::tbegin(row_index_));
  }
}

// Sample a DMatrix based on the given gradient pairs.
GradientBasedSample GradientBasedSampler::Sample(common::Span<GradientPair> gpair,
                                                 DMatrix* dmat) {
  monitor_.StartCuda("Sample");
  GradientBasedSample sample;
  if (is_sampling_) {
    switch (sampling_method_) {
      case TrainParam::kUniform:
        sample = UniformSampling(gpair, dmat);
        break;
      case TrainParam::kGradientBased:
        sample = GradientBasedSampling(gpair, dmat);
        break;
      default:
        LOG(FATAL) << "unknown sampling method";
        sample = {0, nullptr, gpair};
    }
  } else {
    sample = NoSampling(gpair, dmat);
  }
  monitor_.StopCuda("Sample");
  return sample;
}

GradientBasedSample GradientBasedSampler::NoSampling(common::Span<GradientPair> gpair,
                                                     DMatrix* dmat) {
  if (is_external_memory_) {
    ConcatenatePages(dmat);
    return {dmat->Info().num_row_, page_.get(), gpair};
  } else {
    return {dmat->Info().num_row_, original_page_, gpair};
  }
}

// When not sampling, concatenate all the external memory ELLPACK pages into a single in-memory
// page.
void GradientBasedSampler::ConcatenatePages(DMatrix* dmat) {
  if (page_concatenated_) {
    return;
  }

  size_t offset = 0;
  for (auto& batch : dmat->GetBatches<EllpackPage>(batch_param_)) {
    auto page = batch.Impl();
    size_t num_elements = page_->Copy(batch_param_.gpu_id, page, offset);
    offset += num_elements;
  }
  page_concatenated_ = true;
}

/*! \brief A functor that returns random weights. */
struct RandomWeight : public thrust::unary_function<size_t, float> {
  uint32_t seed;

  XGBOOST_DEVICE explicit RandomWeight(size_t _seed) : seed(_seed) {}

  XGBOOST_DEVICE float operator()(size_t i) const {
    thrust::default_random_engine rng(seed);
    thrust::uniform_real_distribution<float> dist;
    rng.discard(i);
    return dist(rng);
  }
};

/*! \brief A functor that scales gradient pairs by 1/p. */
struct FixedScaling : public thrust::unary_function<GradientPair, GradientPair> {
  float p;

  XGBOOST_DEVICE explicit FixedScaling(float _p) : p(_p) {}

  XGBOOST_DEVICE GradientPair operator()(const GradientPair& gpair) const {
    return gpair / p;
  }
};

GradientBasedSample GradientBasedSampler::UniformSampling(common::Span<GradientPair> gpair,
                                                          DMatrix* dmat) {
  // Generate random weights.
  thrust::transform(thrust::counting_iterator<size_t>(0),
                    thrust::counting_iterator<size_t>(gpair.size()),
                    dh::tbegin(row_weight_),
                    RandomWeight(common::GlobalRandom()()));
  // Scale gradient pairs by 1/subsample.
  thrust::transform(dh::tbegin(gpair), dh::tend(gpair),
                    dh::tbegin(gpair),
                    FixedScaling(subsample_));
  return SequentialPoissonSampling(gpair, dmat);
}

/*! \brief A functor that combines the gradient pair into a single float.
 *
 * The approach here is based on Minimal Variance Sampling (MVS), with lambda set to 0.1.
 *
 * \see Ibragimov, B., & Gusev, G. (2019). Minimal Variance Sampling in Stochastic Gradient
 * Boosting. In Advances in Neural Information Processing Systems (pp. 15061-15071).
 */
struct CombineGradientPair : public thrust::unary_function<GradientPair, float> {
  static constexpr float kLambda = 0.1f;

  XGBOOST_DEVICE float operator()(const GradientPair& gpair) const {
    return sqrtf(powf(gpair.GetGrad(), 2) + kLambda * powf(gpair.GetHess(), 2));
  }
};

/*! \brief A functor that calculates the weight of each row, and scales gradient pairs by 1/p_i. */
struct CalculateWeight
    : public thrust::binary_function<GradientPair, size_t, thrust::tuple<float, GradientPair>> {
  size_t sample_rows;
  float normalization;
  RandomWeight rnd;
  CombineGradientPair combine;

  XGBOOST_DEVICE CalculateWeight(size_t _sample_rows, float _normalization, RandomWeight _rnd)
      : sample_rows(_sample_rows), normalization(_normalization), rnd(_rnd) {}

  XGBOOST_DEVICE thrust::tuple<float, GradientPair> operator()(const GradientPair& gpair,
                                                               size_t i) {
    // If the gradient and hessian are both empty, we should never select this row.
    if (gpair.GetGrad() == 0 && gpair.GetHess() == 0) {
      return thrust::make_tuple(FLT_MAX, gpair);
    }
    float combined_gradient = combine(gpair);
    float p = sample_rows * combined_gradient / normalization;
    if (p >= 1) {
      // Always select this row.
      return thrust::make_tuple(0.0f, gpair);
    } else {
      // Select this row randomly with probability proportional to the combined gradient.
      // Scale gpair by 1/p.
      return thrust::make_tuple(rnd(i) / combined_gradient, gpair / p);
    }
  }
};

GradientBasedSample GradientBasedSampler::GradientBasedSampling(
    common::Span<GradientPair> gpair, DMatrix* dmat) {
  float normalization = thrust::transform_reduce(dh::tbegin(gpair), dh::tend(gpair),
                                                 CombineGradientPair(),
                                                 0.0f,
                                                 thrust::plus<float>());
  thrust::transform(dh::tbegin(gpair), dh::tend(gpair),
                    thrust::counting_iterator<size_t>(0),
                    thrust::make_zip_iterator(thrust::make_tuple(
                        dh::tbegin(row_weight_), dh::tbegin(gpair))),
                    CalculateWeight(sample_rows_, normalization,
                        RandomWeight(common::GlobalRandom()())));
  return SequentialPoissonSampling(gpair, dmat);
}

/*! \brief A functor that returns true if the gradient pair is non-zero. */
struct IsNonZero : public thrust::unary_function<GradientPair, bool> {
  XGBOOST_DEVICE bool operator()(const GradientPair& gpair) const {
    return gpair.GetGrad() != 0 || gpair.GetHess() != 0;
  }
};

/*! \brief A functor that clears the row indexes with empty gradient. */
struct ClearEmptyRows : public thrust::binary_function<GradientPair, size_t, size_t> {
  XGBOOST_DEVICE size_t operator()(const GradientPair& gpair, size_t row_index) const {
    if (gpair.GetGrad() != 0 || gpair.GetHess() != 0) {
      return row_index;
    } else {
      return SIZE_MAX;
    }
  }
};

// Perform sampling after the weights are calculated.
GradientBasedSample GradientBasedSampler::SequentialPoissonSampling(
      common::Span<GradientPair> gpair, DMatrix* dmat) {
  // Sort the gradient pairs and row indexes by weight.
  thrust::sort_by_key(dh::tbegin(row_weight_), dh::tend(row_weight_),
                      thrust::make_zip_iterator(thrust::make_tuple(dh::tbegin(gpair),
                                                                   dh::tbegin(row_index_))));

  // Clear the gradient pairs not included in the sample.
  thrust::fill(dh::tbegin(gpair) + sample_rows_, dh::tend(gpair), GradientPair());

  // Mask the sample rows.
  thrust::fill(dh::tbegin(sample_row_index_), dh::tbegin(sample_row_index_) + sample_rows_, 1);
  thrust::fill(dh::tbegin(sample_row_index_) + sample_rows_, dh::tend(sample_row_index_), 0);

  // Sort the gradient pairs and sample row indexes by the original row index.
  thrust::sort_by_key(dh::tbegin(row_index_), dh::tend(row_index_),
                      thrust::make_zip_iterator(thrust::make_tuple(dh::tbegin(gpair),
                                                                   dh::tbegin(sample_row_index_))));

  // Compact the non-zero gradient pairs.
  thrust::copy_if(dh::tbegin(gpair), dh::tend(gpair), dh::tbegin(gpair_), IsNonZero());

  // Index the sample rows.
  thrust::exclusive_scan(dh::tbegin(sample_row_index_), dh::tend(sample_row_index_),
                         dh::tbegin(sample_row_index_));
  thrust::transform(dh::tbegin(gpair), dh::tend(gpair),
                    dh::tbegin(sample_row_index_),
                    dh::tbegin(sample_row_index_),
                    ClearEmptyRows());

  // Compact the ELLPACK pages into the single sample page.
  thrust::fill(dh::tbegin(page_->gidx_buffer), dh::tend(page_->gidx_buffer), 0);
  for (auto& batch : dmat->GetBatches<EllpackPage>(batch_param_)) {
    page_->Compact(batch_param_.gpu_id, batch.Impl(), sample_row_index_);
  }

  return {sample_rows_, page_.get(), gpair_};
}

};  // namespace tree
};  // namespace xgboost
