#include "hip/hip_runtime.h"
/*!
 * Copyright 2019 by XGBoost Contributors
 */
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <xgboost/host_device_vector.h>
#include <xgboost/logging.h>

#include <algorithm>

#include "../../common/compressed_iterator.h"
#include "../../common/random.h"
#include "gradient_based_sampler.cuh"

namespace xgboost {
namespace tree {

GradientBasedSampler::GradientBasedSampler(BatchParam batch_param,
                                           EllpackInfo info,
                                           size_t n_rows,
                                           float subsample,
                                           SamplingMethod sampling_method)
    : batch_param_(batch_param), info_(info), sampling_method_(sampling_method) {
  monitor_.Init("gradient_based_sampler");

  if (subsample == 0.0f || subsample == 1.0f) {
    sample_rows_ = MaxSampleRows(n_rows);
  } else {
    sample_rows_ = n_rows * subsample;
  }

  if (sample_rows_ >= n_rows) {
    sampling_method_ = kNoSampling;
    sample_rows_ = n_rows;
    LOG(CONSOLE) << "Keeping " << sample_rows_ << " rows in GPU memory, not sampling";
  } else {
    LOG(CONSOLE) << "Sampling " << sample_rows_ << " rows";
  }

  page_.reset(new EllpackPageImpl(batch_param.gpu_id, info, sample_rows_));
  if (sampling_method_ != kNoSampling) {
    ba_.Allocate(batch_param_.gpu_id,
                 &gpair_, sample_rows_,
                 &row_weight_, n_rows,
                 &row_index_, n_rows,
                 &sample_row_index_, n_rows);
    thrust::copy(thrust::counting_iterator<size_t>(0),
                 thrust::counting_iterator<size_t>(n_rows),
                 dh::tbegin(row_index_));
  }
}

size_t GradientBasedSampler::MaxSampleRows(size_t n_rows) {
  size_t available_memory = dh::AvailableMemory(batch_param_.gpu_id);
  // Subtract row_weight_, row_index_, and sample_row_index_.
  available_memory -= n_rows * (sizeof(float) + 2 * sizeof(size_t));
  size_t usable_memory = available_memory * 0.7;
  size_t extra_bytes = sizeof(GradientPair);
  size_t max_rows = common::CompressedBufferWriter::CalculateMaxRows(
      usable_memory, info_.NumSymbols(), info_.row_stride, extra_bytes);
  return max_rows;
}

GradientBasedSample GradientBasedSampler::Sample(common::Span<GradientPair> gpair,
                                                 DMatrix* dmat) {
  switch (sampling_method_) {
    case kNoSampling:
      return NoSampling(gpair, dmat);
    case kSequentialPoissonSampling:
      return SequentialPoissonSampling(gpair, dmat);
    case kUniformSampling:
      return UniformSampling(gpair, dmat);
    default:
      LOG(FATAL) << "unknown sampling method";
      return {sample_rows_, page_.get(), gpair};
  }
}

void GradientBasedSampler::CollectPages(DMatrix* dmat) {
  if (page_collected_) {
    return;
  }

  size_t offset = 0;
  for (auto& batch : dmat->GetBatches<EllpackPage>(batch_param_)) {
    auto page = batch.Impl();
    size_t num_elements = page_->Copy(batch_param_.gpu_id, page, offset);
    offset += num_elements;
  }
  page_collected_ = true;
}

GradientBasedSample GradientBasedSampler::NoSampling(common::Span<GradientPair> gpair,
                                                     DMatrix* dmat) {
  CollectPages(dmat);
  return {sample_rows_, page_.get(), gpair};
}

/*! \brief A functor that calculate the weight of each row as random(0, 1) / abs(grad). */
struct CalculateWeight : public thrust::binary_function<GradientPair, size_t, float> {
  const uint32_t seed;

  XGBOOST_DEVICE explicit CalculateWeight(size_t _seed) : seed(_seed) {}

  XGBOOST_DEVICE float operator()(const GradientPair& gpair, size_t i) {
    if (gpair.GetGrad() == 0) {
      return FLT_MAX;
    }
    thrust::default_random_engine rng(seed);
    thrust::uniform_real_distribution<float> dist;
    rng.discard(i);
    return dist(rng) / fabsf(gpair.GetGrad());
  }
};

/*! \brief A functor that returns true if the gradient pair is non-zero. */
struct IsNonZero : public thrust::unary_function<GradientPair, bool> {
  XGBOOST_DEVICE bool operator()(const GradientPair& gpair) const {
    return gpair.GetGrad() != 0 || gpair.GetHess() != 0;
  }
};

/*! \brief A functor that clears the row indexes with empty gradient. */
struct ClearEmptyRows : public thrust::binary_function<GradientPair, size_t, size_t> {
  const size_t max_rows;

  XGBOOST_DEVICE explicit ClearEmptyRows(size_t max_rows) : max_rows(max_rows) {}

  XGBOOST_DEVICE size_t operator()(const GradientPair& gpair, size_t row_index) const {
    if ((gpair.GetGrad() != 0 || gpair.GetHess() != 0) && row_index < max_rows) {
      return row_index;
    } else {
      return SIZE_MAX;
    }
  }
};

GradientBasedSample GradientBasedSampler::SequentialPoissonSampling(
    common::Span<xgboost::GradientPair> gpair, DMatrix* dmat) {
  // Transform the gradient to weight = random(0, 1) / abs(grad).
  thrust::transform(dh::tbegin(gpair), dh::tend(gpair),
                    thrust::counting_iterator<size_t>(0),
                    dh::tbegin(row_weight_),
                    CalculateWeight(common::GlobalRandom()()));
  return WeightedSampling(gpair, dmat);
}

GradientBasedSample GradientBasedSampler::WeightedSampling(
      common::Span<xgboost::GradientPair> gpair, DMatrix* dmat) {
  // Sort the gradient pairs and row indexes by weight.
  thrust::sort_by_key(dh::tbegin(row_weight_), dh::tend(row_weight_),
                      thrust::make_zip_iterator(thrust::make_tuple(dh::tbegin(gpair),
                                                                   dh::tbegin(row_index_))));

  // Clear the gradient pairs not in the sample.
  thrust::fill(dh::tbegin(gpair) + sample_rows_, dh::tend(gpair), GradientPair());

  // Mask the sample rows.
  thrust::fill(dh::tbegin(sample_row_index_), dh::tbegin(sample_row_index_) + sample_rows_, 1);
  thrust::fill(dh::tbegin(sample_row_index_) + sample_rows_, dh::tend(sample_row_index_), 0);

  // Sort the gradient pairs and sample row indexes by the original row index.
  thrust::sort_by_key(dh::tbegin(row_index_), dh::tend(row_index_),
                      thrust::make_zip_iterator(thrust::make_tuple(dh::tbegin(gpair),
                                                                   dh::tbegin(sample_row_index_))));

  // Compact the non-zero gradient pairs.
  thrust::copy_if(dh::tbegin(gpair), dh::tend(gpair), dh::tbegin(gpair_), IsNonZero());

  // Index the sample rows.
  thrust::exclusive_scan(dh::tbegin(sample_row_index_), dh::tend(sample_row_index_),
                         dh::tbegin(sample_row_index_));
  thrust::transform(dh::tbegin(gpair), dh::tend(gpair),
                    dh::tbegin(sample_row_index_),
                    dh::tbegin(sample_row_index_),
                    ClearEmptyRows(sample_rows_));

  // Compact the ELLPACK pages into the single sample page.
  thrust::fill(dh::tbegin(page_->gidx_buffer), dh::tend(page_->gidx_buffer), 0);
  for (auto& batch : dmat->GetBatches<EllpackPage>(batch_param_)) {
    page_->Compact(batch_param_.gpu_id, batch.Impl(), sample_row_index_);
  }

  return {sample_rows_, page_.get(), gpair_};
}

/*! \brief A functor that returns random weights. */
struct RandomWeight : public thrust::unary_function<size_t, float> {
  const uint32_t seed;

  XGBOOST_DEVICE explicit RandomWeight(size_t _seed) : seed(_seed) {}

  XGBOOST_DEVICE float operator()(size_t i) {
    thrust::default_random_engine rng(seed);
    thrust::uniform_real_distribution<float> dist;
    rng.discard(i);
    return dist(rng);
  }
};

GradientBasedSample GradientBasedSampler::UniformSampling(common::Span<GradientPair> gpair,
                                                          DMatrix* dmat) {
  // Generate random weights.
  thrust::transform(thrust::counting_iterator<size_t>(0),
                    thrust::counting_iterator<size_t>(0) + gpair.size(),
                    dh::tbegin(row_weight_),
                    RandomWeight(common::GlobalRandom()()));
  return WeightedSampling(gpair, dmat);
}
};  // namespace tree
};  // namespace xgboost
