
/*!
 * Copyright 2017-2019 XGBoost contributors
 */
#include <thrust/sequence.h>
#include <vector>
#include "../../common/device_helpers.cuh"
#include "row_partitioner.cuh"

namespace xgboost {
namespace tree {

struct IndicateLeftTransform {
  RowPartitioner::TreePositionT left_nidx;
  explicit IndicateLeftTransform(RowPartitioner::TreePositionT left_nidx)
      : left_nidx(left_nidx) {}
  __host__ __device__ __forceinline__ int operator()(
      const RowPartitioner::TreePositionT& x) const {
    return x == left_nidx ? 1 : 0;
  }
};

void RowPartitioner::SortPosition(common::Span<TreePositionT> position,
                                  common::Span<TreePositionT> position_out,
                                  common::Span<RowIndexT> ridx,
                                  common::Span<RowIndexT> ridx_out,
                                  TreePositionT left_nidx,
                                  TreePositionT right_nidx,
                                  int64_t* d_left_count, hipStream_t stream) {
  auto d_position_out = position_out.data();
  auto d_position_in = position.data();
  auto d_ridx_out = ridx_out.data();
  auto d_ridx_in = ridx.data();
  auto write_results = [=] __device__(size_t idx, int ex_scan_result) {
    int scatter_address;
    if (d_position_in[idx] == left_nidx) {
      scatter_address = ex_scan_result;
    } else {
      scatter_address = (idx - ex_scan_result) + *d_left_count;
    }
    d_position_out[scatter_address] = d_position_in[idx];
    d_ridx_out[scatter_address] = d_ridx_in[idx];
  };  // NOLINT

  IndicateLeftTransform conversion_op(left_nidx);
  hipcub::TransformInputIterator<TreePositionT, IndicateLeftTransform,
                              TreePositionT*>
      in_itr(d_position_in, conversion_op);
  dh::DiscardLambdaItr<decltype(write_results)> out_itr(write_results);
  size_t temp_storage_bytes = 0;
  hipcub::DeviceScan::ExclusiveSum(nullptr, temp_storage_bytes, in_itr, out_itr,
                                position.size(), stream);
  dh::caching_device_vector<uint8_t> temp_storage(temp_storage_bytes);
  hipcub::DeviceScan::ExclusiveSum(temp_storage.data().get(), temp_storage_bytes,
                                in_itr, out_itr, position.size(), stream);
}
RowPartitioner::RowPartitioner(int device_idx, size_t num_rows)
    : device_idx(device_idx) {
  dh::safe_cuda(hipSetDevice(device_idx));
  ridx_a.resize(num_rows);
  ridx_b.resize(num_rows);
  position_a.resize(num_rows);
  position_b.resize(num_rows);
  ridx = dh::DoubleBuffer<RowIndexT>{&ridx_a, &ridx_b};
  position = dh::DoubleBuffer<TreePositionT>{&position_a, &position_b};
  ridx_segments.emplace_back(Segment(0, num_rows));

  thrust::sequence(
      thrust::device_pointer_cast(ridx.CurrentSpan().data()),
      thrust::device_pointer_cast(ridx.CurrentSpan().data() + ridx.Size()));
  thrust::fill(
      thrust::device_pointer_cast(position.Current()),
      thrust::device_pointer_cast(position.Current() + position.Size()), 0);
  left_counts.resize(256);
  thrust::fill(left_counts.begin(), left_counts.end(), 0);
  streams.resize(2);
  for (auto& stream : streams) {
    dh::safe_cuda(hipStreamCreate(&stream));
  }
}
RowPartitioner::~RowPartitioner() {
  dh::safe_cuda(hipSetDevice(device_idx));
  for (auto& stream : streams) {
    dh::safe_cuda(hipStreamDestroy(stream));
  }
}

common::Span<const RowPartitioner::RowIndexT> RowPartitioner::GetRows(
    TreePositionT nidx) {
  auto segment = ridx_segments.at(nidx);
  // Return empty span here as a valid result
  // Will error if we try to construct a span from a pointer with size 0
  if (segment.Size() == 0) {
    return common::Span<const RowPartitioner::RowIndexT>();
  }
  return ridx.CurrentSpan().subspan(segment.begin, segment.Size());
}

common::Span<const RowPartitioner::RowIndexT> RowPartitioner::GetRows() {
  return ridx.CurrentSpan();
}

common::Span<const RowPartitioner::TreePositionT>
RowPartitioner::GetPosition() {
  return position.CurrentSpan();
}
std::vector<RowPartitioner::RowIndexT> RowPartitioner::GetRowsHost(
    TreePositionT nidx) {
  auto span = GetRows(nidx);
  std::vector<RowIndexT> rows(span.size());
  dh::CopyDeviceSpanToVector(&rows, span);
  return rows;
}

std::vector<RowPartitioner::TreePositionT> RowPartitioner::GetPositionHost() {
  auto span = GetPosition();
  std::vector<TreePositionT> position(span.size());
  dh::CopyDeviceSpanToVector(&position, span);
  return position;
}

void RowPartitioner::SortPositionAndCopy(const Segment& segment,
                                         TreePositionT left_nidx,
                                         TreePositionT right_nidx,
                                         int64_t* d_left_count,
                                         hipStream_t stream) {
  SortPosition(
      common::Span<TreePositionT>(position.Current() + segment.begin,
                                  segment.Size()),
      common::Span<TreePositionT>(position.other() + segment.begin,
                                  segment.Size()),
      common::Span<RowIndexT>(ridx.Current() + segment.begin, segment.Size()),
      common::Span<RowIndexT>(ridx.other() + segment.begin, segment.Size()),
      left_nidx, right_nidx, d_left_count, stream);
  // Copy back key/value
  const auto d_position_current = position.Current() + segment.begin;
  const auto d_position_other = position.other() + segment.begin;
  const auto d_ridx_current = ridx.Current() + segment.begin;
  const auto d_ridx_other = ridx.other() + segment.begin;
  dh::LaunchN(device_idx, segment.Size(), stream, [=] __device__(size_t idx) {
    d_position_current[idx] = d_position_other[idx];
    d_ridx_current[idx] = d_ridx_other[idx];
  });
}
};  // namespace tree
};  // namespace xgboost
