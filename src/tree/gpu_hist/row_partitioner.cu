/*!
 * Copyright 2017-2022 XGBoost contributors
 */
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/sequence.h>

#include <vector>

#include "../../common/device_helpers.cuh"
#include "row_partitioner.cuh"

namespace xgboost {
namespace tree {

RowPartitioner::RowPartitioner(int device_idx, size_t num_rows)
    : device_idx_(device_idx), ridx_(num_rows), ridx_tmp_(num_rows) {
  dh::safe_cuda(hipSetDevice(device_idx_));
  ridx_segments_.emplace_back(NodePositionInfo{Segment(0, num_rows)});
  thrust::sequence(thrust::device, ridx_.data(), ridx_.data() + ridx_.size());
  dh::safe_cuda(hipStreamCreate(&stream_));
}

RowPartitioner::~RowPartitioner() {
  dh::safe_cuda(hipSetDevice(device_idx_));
  dh::safe_cuda(hipStreamDestroy(stream_));
}

common::Span<const RowPartitioner::RowIndexT> RowPartitioner::GetRows(bst_node_t nidx) {
  auto segment = ridx_segments_.at(nidx).segment;
  return dh::ToSpan(ridx_).subspan(segment.begin, segment.Size());
}

common::Span<const RowPartitioner::RowIndexT> RowPartitioner::GetRows() {
  return dh::ToSpan(ridx_);
}

std::vector<RowPartitioner::RowIndexT> RowPartitioner::GetRowsHost(bst_node_t nidx) {
  auto span = GetRows(nidx);
  std::vector<RowIndexT> rows(span.size());
  dh::CopyDeviceSpanToVector(&rows, span);
  return rows;
}

};  // namespace tree
};  // namespace xgboost
