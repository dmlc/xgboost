#include "hip/hip_runtime.h"
/*!
 * Copyright 2020 by XGBoost Contributors
 */
#include <thrust/reduce.h>
#include <thrust/iterator/transform_iterator.h>
#include <algorithm>
#include <ctgmath>
#include <limits>

#include "xgboost/base.h"
#include "row_partitioner.cuh"

#include "histogram.cuh"

#include "../../data/ellpack_page.cuh"
#include "../../common/device_helpers.cuh"

namespace xgboost {
namespace tree {
// Following 2 functions are slightly modified version of fbcuda.

/* \brief Constructs a rounding factor used to truncate elements in a sum such that the
   sum of the truncated elements is the same no matter what the order of the sum is.

 * Algorithm 5: Reproducible Sequential Sum in 'Fast Reproducible Floating-Point
 * Summation' by Demmel and Nguyen

 * In algorithm 5 the bound is calculated as $max(|v_i|) * n$.  Here we use the bound
 *
 * \begin{equation}
 *   max( fl(\sum^{V}_{v_i>0}{v_i}), fl(\sum^{V}_{v_i<0}|v_i|) )
 * \end{equation}
 *
 * to avoid outliers, as the full reduction is reproducible on GPU with reduction tree.
 */
template <typename T>
XGBOOST_DEV_INLINE __host__ T CreateRoundingFactor(T max_abs, int n) {
  T delta = max_abs / (static_cast<T>(1.0) - 2 * n * std::numeric_limits<T>::epsilon());

  // Calculate ceil(log_2(delta)).
  // frexpf() calculates exp and returns `x` such that
  // delta = x * 2^exp, where `x` in (-1.0, -0.5] U [0.5, 1).
  // Because |x| < 1, exp is exactly ceil(log_2(delta)).
  int exp;
  std::frexp(delta, &exp);

  // return M = 2 ^ ceil(log_2(delta))
  return std::ldexp(static_cast<T>(1.0), exp);
}

namespace {
struct Pair {
  GradientPair first;
  GradientPair second;
};
__host__ XGBOOST_DEV_INLINE Pair operator+(Pair const& lhs, Pair const& rhs) {
  return {lhs.first + rhs.first, lhs.second + rhs.second};
}
}  // anonymous namespace

struct Clip : public thrust::unary_function<GradientPair, Pair> {
  static XGBOOST_DEV_INLINE float Pclip(float v) {
    return v > 0 ? v : 0;
  }
  static XGBOOST_DEV_INLINE float Nclip(float v) {
    return v < 0 ? abs(v) : 0;
  }

  XGBOOST_DEV_INLINE Pair operator()(GradientPair x) const {
    auto pg = Pclip(x.GetGrad());
    auto ph = Pclip(x.GetHess());

    auto ng = Nclip(x.GetGrad());
    auto nh = Nclip(x.GetHess());

    return { GradientPair{ pg, ph }, GradientPair{ ng, nh } };
  }
};

template <typename GradientSumT>
GradientSumT CreateRoundingFactor(common::Span<GradientPair const> gpair) {
  using T = typename GradientSumT::ValueT;
  dh::XGBCachingDeviceAllocator<char> alloc;

  thrust::device_ptr<GradientPair const> gpair_beg {gpair.data()};
  thrust::device_ptr<GradientPair const> gpair_end {gpair.data() + gpair.size()};
  auto beg = thrust::make_transform_iterator(gpair_beg, Clip());
  auto end = thrust::make_transform_iterator(gpair_end, Clip());
  Pair p = dh::Reduce(thrust::cuda::par(alloc), beg, end, Pair{}, thrust::plus<Pair>{});
  GradientPair positive_sum {p.first}, negative_sum {p.second};

  auto histogram_rounding = GradientSumT {
    CreateRoundingFactor<T>(std::max(positive_sum.GetGrad(), negative_sum.GetGrad()),
                            gpair.size()),
    CreateRoundingFactor<T>(std::max(positive_sum.GetHess(), negative_sum.GetHess()),
                            gpair.size()) };
  return histogram_rounding;
}

template GradientPairPrecise CreateRoundingFactor(common::Span<GradientPair const> gpair);
template GradientPair CreateRoundingFactor(common::Span<GradientPair const> gpair);

template <typename GradientSumT, bool use_shared_memory_histograms>
__global__ void SharedMemHistKernel(EllpackDeviceAccessor matrix,
                                    FeatureGroupsAccessor feature_groups,
                                    common::Span<const RowPartitioner::RowIndexT> d_ridx,
                                    GradientSumT* __restrict__ d_node_hist,
                                    const GradientPair* __restrict__ d_gpair,
                                    GradientSumT const rounding,
                                    GradientSumT adjust_rounding,
                                    GradientSumT inv_adjust_rounding) {
  using T = typename GradientSumT::ValueT;
  extern __shared__ char smem[];
  FeatureGroup group = feature_groups[blockIdx.y];
  GradientPairInt32 *smem_arr = reinterpret_cast<GradientPairInt32 *>(smem);
  if (use_shared_memory_histograms) {
    dh::BlockFill(smem_arr, group.num_bins, GradientPairInt32());
    __syncthreads();
  }
  int feature_stride = matrix.is_dense ? group.num_features : matrix.row_stride;
  size_t n_elements = feature_stride * d_ridx.size();
  for (auto idx : dh::GridStrideRange(static_cast<size_t>(0), n_elements)) {
    int ridx = d_ridx[idx / feature_stride];
    int gidx = matrix.gidx_iter[ridx * matrix.row_stride + group.start_feature +
                                idx % feature_stride];
    if (gidx != matrix.NumBins()) {
      // If we are not using shared memory, accumulate the values directly into
      // global memory
      gidx = use_shared_memory_histograms ? gidx - group.start_bin : gidx;
      if (use_shared_memory_histograms) {
        auto adjusted = GradientPairInt32(
            d_gpair[ridx].GetGrad() * ::round(inv_adjust_rounding.GetGrad()),
            d_gpair[ridx].GetHess() * ::round(inv_adjust_rounding.GetHess()));
        dh::AtomicAddGpair(smem_arr + gidx, adjusted);
      } else {
        GradientSumT truncated{
            TruncateWithRoundingFactor<T>(rounding.GetGrad(),
                                          d_gpair[ridx].GetGrad()),
            TruncateWithRoundingFactor<T>(rounding.GetHess(),
                                          d_gpair[ridx].GetHess()),
        };
        dh::AtomicAddGpair(d_node_hist + gidx, truncated);
      }
    }
  }

  if (use_shared_memory_histograms) {
    // Write shared memory back to global memory
    __syncthreads();
    for (auto i : dh::BlockStrideRange(0, group.num_bins)) {
      GradientSumT sum =
          GradientSumT(smem_arr[i].GetGrad() * adjust_rounding.GetGrad(),
                       smem_arr[i].GetHess() * adjust_rounding.GetHess());
      GradientSumT truncated{
          TruncateWithRoundingFactor<T>(rounding.GetGrad(), sum.GetGrad()),
          TruncateWithRoundingFactor<T>(rounding.GetHess(), sum.GetHess()),
      };
      dh::AtomicAddGpair(d_node_hist + group.start_bin + i, truncated);
    }
  }
}

std::string floatToBinary(float f) {
  union {
    float f;
    uint32_t i;
  } u;
  u.f = f;
  std::string str;

  for (int i = 0; i < 32; i++) {
    if (u.i % 2) {
      str.push_back('1');
    }
    else {
      str.push_back('0');
    }
    u.i >>= 1;
  }

  // Reverse the string since now it's backwards
  std::string temp(str.rbegin(), str.rend());
  return temp;
}

struct FixedPoint {
  uint32_t value;
};

FixedPoint XGBOOST_DEVICE Float2Fix(float input) {
  FixedPoint v;
  v.value = ::round(input * (1u << 30));
  return v;
}

float FixedToFloat(FixedPoint value) {
  auto v = float(value.value) / float(1u << 30);
  return v;
}

template <typename GradientSumT>
void BuildGradientHistogram(EllpackDeviceAccessor const& matrix,
                            FeatureGroupsAccessor const& feature_groups,
                            common::Span<GradientPair const> gpair,
                            common::Span<const uint32_t> d_ridx,
                            common::Span<GradientSumT> histogram,
                            GradientSumT rounding) {
  // decide whether to use shared memory
  int device = 0;
  dh::safe_cuda(hipGetDevice(&device));
  int max_shared_memory = dh::MaxSharedMemoryOptin(device);
  //size_t smem_size = sizeof(GradientSumT) * feature_groups.max_group_bins;
  size_t smem_size = sizeof(GradientPairInt32) * feature_groups.max_group_bins;
  bool shared = smem_size <= max_shared_memory;
  smem_size = shared ? smem_size : 0;

  // opt into maximum shared memory for the kernel if necessary
  // auto kernel = SharedMemHistKernel<GradientSumT, shared>;

  auto runit = [&](auto kernel) {
    if (shared) {
      dh::safe_cuda(hipFuncSetAttribute(reinterpret_cast<const void*>(
          kernel), hipFuncAttributeMaxDynamicSharedMemorySize,
          max_shared_memory));
    }

    // determine the launch configuration
    int min_grid_size;
    int block_threads = 1024;
    dh::safe_cuda(hipOccupancyMaxPotentialBlockSize(
        &min_grid_size, &block_threads, kernel, smem_size, 0));

    int num_groups = feature_groups.NumGroups();
    int n_mps = 0;
    dh::safe_cuda(
        hipDeviceGetAttribute(&n_mps, hipDeviceAttributeMultiprocessorCount, device));
    int n_blocks_per_mp = 0;
    dh::safe_cuda(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &n_blocks_per_mp, kernel, block_threads, smem_size));
    unsigned grid_size = n_blocks_per_mp * n_mps;

    // TODO(canonizer): This is really a hack, find a better way to distribute
    // the data among thread blocks. The intention is to generate enough thread
    // blocks to fill the GPU, but avoid having too many thread blocks, as this
    // is less efficient when the number of rows is low. At least one thread
    // block per feature group is required. The number of thread blocks:
    // - for num_groups <= num_groups_threshold, around  grid_size * num_groups
    // - for num_groups_threshold <= num_groups <= num_groups_threshold *
    // grid_size,
    //     around grid_size * num_groups_threshold
    // - for num_groups_threshold * grid_size <= num_groups, around num_groups
    int num_groups_threshold = 4;
    grid_size = common::DivRoundUp(
        grid_size, common::DivRoundUp(num_groups, num_groups_threshold));

    using T = typename GradientSumT::ValueT;
    GradientSumT adjust_rounding = rounding / T(1 << 30); // keep 1 for sign bit
    GradientSumT inv_adjust_rounding = GradientSumT(
        T(1) / adjust_rounding.GetGrad(), T(1) / adjust_rounding.GetHess());
    dh::LaunchKernel{dim3(grid_size, num_groups),
                     static_cast<uint32_t>(block_threads), smem_size}(
        kernel, matrix, feature_groups, d_ridx, histogram.data(), gpair.data(),
        rounding, adjust_rounding, inv_adjust_rounding);
  };
  if (shared) {
    runit(SharedMemHistKernel<GradientSumT, true>);
  } else {
    runit(SharedMemHistKernel<GradientSumT, false>);
  }

  dh::safe_cuda(hipGetLastError());
}

template void BuildGradientHistogram<GradientPair>(
    EllpackDeviceAccessor const& matrix,
    FeatureGroupsAccessor const& feature_groups,
    common::Span<GradientPair const> gpair,
    common::Span<const uint32_t> ridx,
    common::Span<GradientPair> histogram,
    GradientPair rounding);

template void BuildGradientHistogram<GradientPairPrecise>(
    EllpackDeviceAccessor const& matrix,
    FeatureGroupsAccessor const& feature_groups,
    common::Span<GradientPair const> gpair,
    common::Span<const uint32_t> ridx,
    common::Span<GradientPairPrecise> histogram,
    GradientPairPrecise rounding);

}  // namespace tree
}  // namespace xgboost
