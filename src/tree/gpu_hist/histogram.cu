#include "hip/hip_runtime.h"
/*!
 * Copyright 2020-2021 by XGBoost Contributors
 */
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>

#include <algorithm>
#include <ctgmath>
#include <limits>

#include "../../common/device_helpers.cuh"
#include "../../data/ellpack_page.cuh"
#include "histogram.cuh"
#include "row_partitioner.cuh"
#include "xgboost/base.h"

namespace xgboost {
namespace tree {
// Following 2 functions are slightly modified version of fbcuda.

/* \brief Constructs a rounding factor used to truncate elements in a sum such that the
   sum of the truncated elements is the same no matter what the order of the sum is.

 * Algorithm 5: Reproducible Sequential Sum in 'Fast Reproducible Floating-Point
 * Summation' by Demmel and Nguyen

 * In algorithm 5 the bound is calculated as $max(|v_i|) * n$.  Here we use the bound
 *
 * \begin{equation}
 *   max( fl(\sum^{V}_{v_i>0}{v_i}), fl(\sum^{V}_{v_i<0}|v_i|) )
 * \end{equation}
 *
 * to avoid outliers, as the full reduction is reproducible on GPU with reduction tree.
 */
template <typename T>
T CreateRoundingFactor(T max_abs, int n) {
  T delta = max_abs / (static_cast<T>(1.0) - 2 * n * std::numeric_limits<T>::epsilon());

  // Calculate ceil(log_2(delta)).
  // frexpf() calculates exp and returns `x` such that
  // delta = x * 2^exp, where `x` in (-1.0, -0.5] U [0.5, 1).
  // Because |x| < 1, exp is exactly ceil(log_2(delta)).
  int exp;
  std::frexp(delta, &exp);

  // return M = 2 ^ ceil(log_2(delta))
  return std::ldexp(static_cast<T>(1.0), exp);
}

namespace {
struct Pair {
  GradientPair first;
  GradientPair second;
};
__host__ XGBOOST_DEV_INLINE Pair operator+(Pair const& lhs, Pair const& rhs) {
  return {lhs.first + rhs.first, lhs.second + rhs.second};
}
}  // anonymous namespace

struct Clip : public thrust::unary_function<GradientPair, Pair> {
  static XGBOOST_DEV_INLINE float Pclip(float v) { return v > 0 ? v : 0; }
  static XGBOOST_DEV_INLINE float Nclip(float v) { return v < 0 ? abs(v) : 0; }

  XGBOOST_DEV_INLINE Pair operator()(GradientPair x) const {
    auto pg = Pclip(x.GetGrad());
    auto ph = Pclip(x.GetHess());

    auto ng = Nclip(x.GetGrad());
    auto nh = Nclip(x.GetHess());

    return {GradientPair{pg, ph}, GradientPair{ng, nh}};
  }
};

GradientQuantiser::GradientQuantiser(common::Span<GradientPair const> gpair) {
  using GradientSumT = GradientPairPrecise;
  using T = typename GradientSumT::ValueT;
  dh::XGBCachingDeviceAllocator<char> alloc;

  thrust::device_ptr<GradientPair const> gpair_beg{gpair.data()};
  auto beg = thrust::make_transform_iterator(gpair_beg, Clip());
  Pair p =
      dh::Reduce(thrust::cuda::par(alloc), beg, beg + gpair.size(), Pair{}, thrust::plus<Pair>{});
  // Treat pair as array of 4 primitive types to allreduce
  using ReduceT = typename decltype(p.first)::ValueT;
  static_assert(sizeof(Pair) == sizeof(ReduceT) * 4, "Expected to reduce four elements.");
  collective::Allreduce<collective::Operation::kSum>(reinterpret_cast<ReduceT*>(&p), 4);
  GradientPair positive_sum{p.first}, negative_sum{p.second};

  std::size_t total_rows = gpair.size();
  collective::Allreduce<collective::Operation::kSum>(&total_rows, 1);

  auto histogram_rounding = GradientSumT{
      CreateRoundingFactor<T>(std::max(positive_sum.GetGrad(), negative_sum.GetGrad()), total_rows),
      CreateRoundingFactor<T>(std::max(positive_sum.GetHess(), negative_sum.GetHess()),
                              total_rows)};

  using IntT = typename GradientPairInt64::ValueT;

  /**
   * Factor for converting gradients from fixed-point to floating-point.
   */
  to_floating_point_ =
      histogram_rounding /
      T(IntT(1) << (sizeof(typename GradientSumT::ValueT) * 8 - 2));  // keep 1 for sign bit
  /**
   * Factor for converting gradients from floating-point to fixed-point. For
   * f64:
   *
   *   Precision = 64 - 1 - log2(rounding)
   *
   * rounding is calcuated as exp(m), see the rounding factor calcuation for
   * details.
   */
  to_fixed_point_ =
      GradientSumT(T(1) / to_floating_point_.GetGrad(), T(1) / to_floating_point_.GetHess());
}


XGBOOST_DEV_INLINE void
AtomicAddGpairShared(xgboost::GradientPairInt64 *dest,
               xgboost::GradientPairInt64 const &gpair) {
  auto dst_ptr = reinterpret_cast<int64_t *>(dest);
  auto g = gpair.GetQuantisedGrad();
  auto h = gpair.GetQuantisedHess();

  AtomicAdd64As32(dst_ptr, g);
  AtomicAdd64As32(dst_ptr + 1, h);
}

// Global 64 bit integer atomics at the time of writing do not benefit from being separated into two
// 32 bit atomics
XGBOOST_DEV_INLINE void AtomicAddGpairGlobal(xgboost::GradientPairInt64* dest,
                                             xgboost::GradientPairInt64 const& gpair) {
  auto dst_ptr = reinterpret_cast<uint64_t*>(dest);
  auto g = gpair.GetQuantisedGrad();
  auto h = gpair.GetQuantisedHess();

  atomicAdd(dst_ptr,
            *reinterpret_cast<uint64_t*>(&g));
  atomicAdd(dst_ptr + 1,
            *reinterpret_cast<uint64_t*>(&h));
}

template <int kBlockThreads, int kItemsPerThread,
          int kItemsPerTile = kBlockThreads* kItemsPerThread>
class HistogramAgent {
  GradientPairInt64* smem_arr_;
  GradientPairInt64* d_node_hist_;
  dh::LDGIterator<const RowPartitioner::RowIndexT> d_ridx_;
  const GradientPair* d_gpair_;
  const FeatureGroup group_;
  const EllpackDeviceAccessor& matrix_;
  const int feature_stride_;
  const std::size_t n_elements_;
  const GradientQuantiser& rounding_;

 public:
  __device__ HistogramAgent(GradientPairInt64* smem_arr,
                            GradientPairInt64* __restrict__ d_node_hist, const FeatureGroup& group,
                            const EllpackDeviceAccessor& matrix,
                            common::Span<const RowPartitioner::RowIndexT> d_ridx,
                            const GradientQuantiser& rounding, const GradientPair* d_gpair)
      : smem_arr_(smem_arr),
        d_node_hist_(d_node_hist),
        d_ridx_(d_ridx.data()),
        group_(group),
        matrix_(matrix),
        feature_stride_(matrix.is_dense ? group.num_features : matrix.row_stride),
        n_elements_(feature_stride_ * d_ridx.size()),
        rounding_(rounding),
        d_gpair_(d_gpair) {}
  __device__ void ProcessPartialTileShared(std::size_t offset) {
    for (std::size_t idx = offset + threadIdx.x;
         idx < min(offset + kBlockThreads * kItemsPerTile, n_elements_); idx += kBlockThreads) {
      int ridx = d_ridx_[idx / feature_stride_];
      int gidx =
          matrix_
              .gidx_iter[ridx * matrix_.row_stride + group_.start_feature + idx % feature_stride_] -
          group_.start_bin;
      if (matrix_.is_dense || gidx != matrix_.NumBins()) {
        auto adjusted = rounding_.ToFixedPoint(d_gpair_[ridx]);
        AtomicAddGpairShared(smem_arr_ + gidx, adjusted);
      }
    }
  }
  // Instruction level parallelism by loop unrolling
  // Allows the kernel to pipeline many operations while waiting for global memory
  // Increases the throughput of this kernel significantly
  __device__ void ProcessFullTileShared(std::size_t offset) {
    std::size_t idx[kItemsPerThread];
    int ridx[kItemsPerThread];
    int gidx[kItemsPerThread];
    GradientPair gpair[kItemsPerThread];
#pragma unroll
    for (int i = 0; i < kItemsPerThread; i++) {
      idx[i] = offset + i * kBlockThreads + threadIdx.x;
    }
#pragma unroll
    for (int i = 0; i < kItemsPerThread; i++) {
      ridx[i] = d_ridx_[idx[i] / feature_stride_];
    }
#pragma unroll
    for (int i = 0; i < kItemsPerThread; i++) {
      gpair[i] = d_gpair_[ridx[i]];
      gidx[i] = matrix_.gidx_iter[ridx[i] * matrix_.row_stride + group_.start_feature +
                                 idx[i] % feature_stride_];
    }
#pragma unroll
    for (int i = 0; i < kItemsPerThread; i++) {
      if ((matrix_.is_dense || gidx[i] != matrix_.NumBins())) {
        auto adjusted = rounding_.ToFixedPoint(gpair[i]);
        AtomicAddGpairShared(smem_arr_ + gidx[i] - group_.start_bin, adjusted);
      }
    }
  }
  __device__ void BuildHistogramWithShared() {
    dh::BlockFill(smem_arr_, group_.num_bins, GradientPairInt64());
    __syncthreads();

    std::size_t offset = blockIdx.x * kItemsPerTile;
    while (offset + kItemsPerTile <= n_elements_) {
      ProcessFullTileShared(offset);
      offset += kItemsPerTile * gridDim.x;
    }
    ProcessPartialTileShared(offset);

    // Write shared memory back to global memory
    __syncthreads();
    for (auto i : dh::BlockStrideRange(0, group_.num_bins)) {
      AtomicAddGpairGlobal(d_node_hist_ + group_.start_bin + i, smem_arr_[i]);
    }
  }

  __device__ void BuildHistogramWithGlobal() {
    for (auto idx : dh::GridStrideRange(static_cast<std::size_t>(0), n_elements_)) {
      int ridx = d_ridx_[idx / feature_stride_];
      int gidx =
          matrix_
              .gidx_iter[ridx * matrix_.row_stride + group_.start_feature + idx % feature_stride_];
      if (matrix_.is_dense || gidx != matrix_.NumBins()) {
        auto adjusted = rounding_.ToFixedPoint(d_gpair_[ridx]);
        AtomicAddGpairGlobal(d_node_hist_ + gidx, adjusted);
      }
    }
  }
};

template <bool use_shared_memory_histograms, int kBlockThreads,
          int kItemsPerThread>
__global__ void __launch_bounds__(kBlockThreads)
    SharedMemHistKernel(const EllpackDeviceAccessor matrix,
                        const FeatureGroupsAccessor feature_groups,
                        common::Span<const RowPartitioner::RowIndexT> d_ridx,
                        GradientPairInt64* __restrict__ d_node_hist,
                        const GradientPair* __restrict__ d_gpair,
                        GradientQuantiser const rounding) {
  extern __shared__ char smem[];
  const FeatureGroup group = feature_groups[blockIdx.y];
  auto smem_arr = reinterpret_cast<GradientPairInt64*>(smem);
  auto agent = HistogramAgent<kBlockThreads, kItemsPerThread>(
      smem_arr, d_node_hist, group, matrix, d_ridx, rounding, d_gpair);
  if (use_shared_memory_histograms) {
    agent.BuildHistogramWithShared();
  } else {
    agent.BuildHistogramWithGlobal();
  }
}

void BuildGradientHistogram(EllpackDeviceAccessor const& matrix,
                            FeatureGroupsAccessor const& feature_groups,
                            common::Span<GradientPair const> gpair,
                            common::Span<const uint32_t> d_ridx,
                            common::Span<GradientPairInt64> histogram,
                            GradientQuantiser rounding, bool force_global_memory) {
  // decide whether to use shared memory
  int device = 0;
  dh::safe_cuda(hipGetDevice(&device));
  // opt into maximum shared memory for the kernel if necessary
  size_t max_shared_memory = dh::MaxSharedMemoryOptin(device);

  size_t smem_size =
      sizeof(GradientPairInt64) * feature_groups.max_group_bins;
  bool shared = !force_global_memory && smem_size <= max_shared_memory;
  smem_size = shared ? smem_size : 0;

  constexpr int kBlockThreads = 1024;
  constexpr int kItemsPerThread = 8;
  constexpr int kItemsPerTile = kBlockThreads * kItemsPerThread;

  auto runit = [&, kMinItemsPerBlock = kItemsPerTile](auto kernel) {
    if (shared) {
      dh::safe_cuda(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize,
                                         max_shared_memory));
    }

    // determine the launch configuration
    int num_groups = feature_groups.NumGroups();
    int n_mps = 0;
    dh::safe_cuda(hipDeviceGetAttribute(&n_mps, hipDeviceAttributeMultiprocessorCount, device));
    int n_blocks_per_mp = 0;
    dh::safe_cuda(hipOccupancyMaxActiveBlocksPerMultiprocessor(&n_blocks_per_mp, kernel,
                                                                kBlockThreads, smem_size));
    // This gives the number of blocks to keep the device occupied
    // Use this as the maximum number of blocks
    unsigned grid_size = n_blocks_per_mp * n_mps;

    // Otherwise launch blocks such that each block has a minimum amount of work to do
    // There are fixed costs to launching each block, e.g. zeroing shared memory
    // The below amount of minimum work was found by experimentation
    int columns_per_group = common::DivRoundUp(matrix.row_stride, feature_groups.NumGroups());
    // Average number of matrix elements processed by each group
    std::size_t items_per_group = d_ridx.size() * columns_per_group;

    // Allocate number of blocks such that each block has about kMinItemsPerBlock work
    // Up to a maximum where the device is saturated
    grid_size =
        min(grid_size,
            unsigned(common::DivRoundUp(items_per_group, kMinItemsPerBlock)));

    dh::LaunchKernel {dim3(grid_size, num_groups),
        static_cast<uint32_t>(kBlockThreads), smem_size}(
        kernel, matrix, feature_groups, d_ridx, histogram.data(), gpair.data(), rounding);
  };

  if (shared) {
    runit(SharedMemHistKernel<true, kBlockThreads, kItemsPerThread>);
  } else {
    runit(SharedMemHistKernel<false, kBlockThreads, kItemsPerThread>);
  }

  dh::safe_cuda(hipGetLastError());
}

}  // namespace tree
}  // namespace xgboost
