#include "hip/hip_runtime.h"
/*!
 * Copyright 2020 by XGBoost Contributors
 */
#include <thrust/reduce.h>
#include <thrust/iterator/transform_iterator.h>
#include <algorithm>
#include <ctgmath>
#include <limits>

#include "xgboost/base.h"
#include "row_partitioner.cuh"

#include "histogram.cuh"

#include "../../data/ellpack_page.cuh"
#include "../../common/device_helpers.cuh"

namespace xgboost {
namespace tree {
// Following 2 functions are slightly modifed version of fbcuda.

/* \brief Constructs a rounding factor used to truncate elements in a sum such that the
   sum of the truncated elements is the same no matter what the order of the sum is.

 * Algorithm 5: Reproducible Sequential Sum in 'Fast Reproducible Floating-Point
 * Summation' by Demmel and Nguyen

 * In algorithm 5 the bound is calculated as $max(|v_i|) * n$.  Here we use the bound
 *
 * \begin{equation}
 *   max( fl(\sum^{V}_{v_i>0}{v_i}), fl(\sum^{V}_{v_i<0}|v_i|) )
 * \end{equation}
 *
 * to avoid outliers, as the full reduction is reproducible on GPU with reduction tree.
 */
template <typename T>
XGBOOST_DEV_INLINE __host__ T CreateRoundingFactor(T max_abs, int n) {
  T delta = max_abs / (static_cast<T>(1.0) - 2 * n * std::numeric_limits<T>::epsilon());

  // Calculate ceil(log_2(delta)).
  // frexpf() calculates exp and returns `x` such that
  // delta = x * 2^exp, where `x` in (-1.0, -0.5] U [0.5, 1).
  // Because |x| < 1, exp is exactly ceil(log_2(delta)).
  int exp;
  std::frexp(delta, &exp);

  // return M = 2 ^ ceil(log_2(delta))
  return std::ldexp(static_cast<T>(1.0), exp);
}

namespace {
struct Pair {
  GradientPair first;
  GradientPair second;
};
__host__ XGBOOST_DEV_INLINE Pair operator+(Pair const& lhs, Pair const& rhs) {
  return {lhs.first + rhs.first, lhs.second + rhs.second};
}
}  // anonymous namespace

struct Clip : public thrust::unary_function<GradientPair, Pair> {
  static XGBOOST_DEV_INLINE float Pclip(float v) {
    return v > 0 ? v : 0;
  }
  static XGBOOST_DEV_INLINE float Nclip(float v) {
    return v < 0 ? abs(v) : 0;
  }

  XGBOOST_DEV_INLINE Pair operator()(GradientPair x) const {
    auto pg = Pclip(x.GetGrad());
    auto ph = Pclip(x.GetHess());

    auto ng = Nclip(x.GetGrad());
    auto nh = Nclip(x.GetHess());

    return { GradientPair{ pg, ph }, GradientPair{ ng, nh } };
  }
};

template <typename GradientSumT>
GradientSumT CreateRoundingFactor(common::Span<GradientPair const> gpair) {
  using T = typename GradientSumT::ValueT;
  dh::XGBCachingDeviceAllocator<char> alloc;

  thrust::device_ptr<GradientPair const> gpair_beg {gpair.data()};
  thrust::device_ptr<GradientPair const> gpair_end {gpair.data() + gpair.size()};
  auto beg = thrust::make_transform_iterator(gpair_beg, Clip());
  auto end = thrust::make_transform_iterator(gpair_end, Clip());
  Pair p = dh::Reduce(thrust::cuda::par(alloc), beg, end, Pair{}, thrust::plus<Pair>{});
  GradientPair positive_sum {p.first}, negative_sum {p.second};

  auto histogram_rounding = GradientSumT {
    CreateRoundingFactor<T>(std::max(positive_sum.GetGrad(), negative_sum.GetGrad()),
                            gpair.size()),
    CreateRoundingFactor<T>(std::max(positive_sum.GetHess(), negative_sum.GetHess()),
                            gpair.size()) };
  return histogram_rounding;
}

template GradientPairPrecise CreateRoundingFactor(common::Span<GradientPair const> gpair);
template GradientPair CreateRoundingFactor(common::Span<GradientPair const> gpair);

template <typename GradientSumT>
__global__ void SharedMemHistKernel(EllpackDeviceAccessor matrix,
                                    FeatureGroupsAccessor feature_groups,
                                    common::Span<const RowPartitioner::RowIndexT> d_ridx,
                                    GradientSumT* __restrict__ d_node_hist,
                                    const GradientPair* __restrict__ d_gpair,
                                    GradientSumT const rounding,
                                    bool use_shared_memory_histograms) {
  using T = typename GradientSumT::ValueT;
  extern __shared__ char smem[];
  FeatureGroup group = feature_groups[blockIdx.y];
  GradientSumT* smem_arr = reinterpret_cast<GradientSumT*>(smem);  // NOLINT
  if (use_shared_memory_histograms) {
    dh::BlockFill(smem_arr, group.num_bins, GradientSumT());
    __syncthreads();
  }
  int feature_stride = matrix.is_dense ? group.num_features : matrix.row_stride;
  size_t n_elements = feature_stride * d_ridx.size();
  for (auto idx : dh::GridStrideRange(static_cast<size_t>(0), n_elements)) {
    int ridx = d_ridx[idx / feature_stride];
    int gidx = matrix.gidx_iter[ridx * matrix.row_stride + group.start_feature +
                                idx % feature_stride];
    if (gidx != matrix.NumBins()) {
      GradientSumT truncated {
        TruncateWithRoundingFactor<T>(rounding.GetGrad(), d_gpair[ridx].GetGrad()),
        TruncateWithRoundingFactor<T>(rounding.GetHess(), d_gpair[ridx].GetHess()),
      };
      // If we are not using shared memory, accumulate the values directly into
      // global memory
      GradientSumT* atomic_add_ptr =
        use_shared_memory_histograms ? smem_arr : d_node_hist;
      gidx = use_shared_memory_histograms ? gidx - group.start_bin : gidx;
      dh::AtomicAddGpair(atomic_add_ptr + gidx, truncated);
    }
  }

  if (use_shared_memory_histograms) {
    // Write shared memory back to global memory
    __syncthreads();
    for (auto i : dh::BlockStrideRange(0, group.num_bins)) {
      GradientSumT truncated{
          TruncateWithRoundingFactor<T>(rounding.GetGrad(),
                                        smem_arr[i].GetGrad()),
          TruncateWithRoundingFactor<T>(rounding.GetHess(),
                                        smem_arr[i].GetHess()),
      };
      dh::AtomicAddGpair(d_node_hist + group.start_bin + i, truncated);
    }
  }
}

template <typename GradientSumT>
void BuildGradientHistogram(EllpackDeviceAccessor const& matrix,
                            FeatureGroupsAccessor const& feature_groups,
                            common::Span<GradientPair const> gpair,
                            common::Span<const uint32_t> d_ridx,
                            common::Span<GradientSumT> histogram,
                            GradientSumT rounding) {
  // decide whether to use shared memory
  int device = 0;
  dh::safe_cuda(hipGetDevice(&device));
  int max_shared_memory = dh::MaxSharedMemoryOptin(device);
  size_t smem_size = sizeof(GradientSumT) * feature_groups.max_group_bins;
  bool shared = smem_size <= max_shared_memory;
  smem_size = shared ? smem_size : 0;

  // opt into maximum shared memory for the kernel if necessary
  auto kernel = SharedMemHistKernel<GradientSumT>;
  if (shared) {
    dh::safe_cuda(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize,
                   max_shared_memory));
  }

  // determine the launch configuration
  int min_grid_size;
  int block_threads = 1024;
  dh::safe_cuda(hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &block_threads, kernel, smem_size, 0));

  int num_groups = feature_groups.NumGroups();
  int n_mps = 0;
  dh::safe_cuda(hipDeviceGetAttribute(&n_mps, hipDeviceAttributeMultiprocessorCount, device));
  int n_blocks_per_mp = 0;
  dh::safe_cuda(hipOccupancyMaxActiveBlocksPerMultiprocessor
                (&n_blocks_per_mp, kernel, block_threads, smem_size));
  unsigned grid_size = n_blocks_per_mp * n_mps;

  // TODO(canonizer): This is really a hack, find a better way to distribute the
  // data among thread blocks.
  // The intention is to generate enough thread blocks to fill the GPU, but
  // avoid having too many thread blocks, as this is less efficient when the
  // number of rows is low. At least one thread block per feature group is
  // required.
  // The number of thread blocks:
  // - for num_groups <= num_groups_threshold, around  grid_size * num_groups
  // - for num_groups_threshold <= num_groups <= num_groups_threshold * grid_size,
  //     around grid_size * num_groups_threshold
  // - for num_groups_threshold * grid_size <= num_groups, around num_groups
  int num_groups_threshold = 4;
  grid_size = common::DivRoundUp(grid_size,
      common::DivRoundUp(num_groups, num_groups_threshold));

  dh::LaunchKernel {
    dim3(grid_size, num_groups), static_cast<uint32_t>(block_threads), smem_size} (
      kernel,
      matrix, feature_groups, d_ridx, histogram.data(), gpair.data(), rounding,
      shared);
  dh::safe_cuda(hipGetLastError());
}

template void BuildGradientHistogram<GradientPair>(
    EllpackDeviceAccessor const& matrix,
    FeatureGroupsAccessor const& feature_groups,
    common::Span<GradientPair const> gpair,
    common::Span<const uint32_t> ridx,
    common::Span<GradientPair> histogram,
    GradientPair rounding);

template void BuildGradientHistogram<GradientPairPrecise>(
    EllpackDeviceAccessor const& matrix,
    FeatureGroupsAccessor const& feature_groups,
    common::Span<GradientPair const> gpair,
    common::Span<const uint32_t> ridx,
    common::Span<GradientPairPrecise> histogram,
    GradientPairPrecise rounding);

}  // namespace tree
}  // namespace xgboost
