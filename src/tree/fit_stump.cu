/**
 * Copyright 2022 by XGBoost Contributors
 *
 * \brief Utilities for estimating initial score.
 */
#if !defined(NOMINMAX) && defined(_WIN32)
#define NOMINMAX
#endif                                            // !defined(NOMINMAX)
#include <thrust/execution_policy.h>              // cuda::par
#include <thrust/iterator/counting_iterator.h>    // thrust::make_counting_iterator

#include <cstddef>                                // std::size_t

#include "../collective/communicator-inl.cuh"
#include "../common/device_helpers.cuh"           // dh::MakeTransformIterator
#include "fit_stump.h"
#include "xgboost/base.h"     // GradientPairPrecise, GradientPair, XGBOOST_DEVICE
#include "xgboost/context.h"  // Context
#include "xgboost/linalg.h"   // TensorView, Tensor, Constant
#include "xgboost/logging.h"  // CHECK_EQ
#include "xgboost/span.h"     // span

namespace xgboost {
namespace tree {
namespace cuda_impl {
void FitStump(Context const* ctx, linalg::TensorView<GradientPair const, 2> gpair,
              linalg::VectorView<float> out) {
  auto n_targets = out.Size();
  CHECK_EQ(n_targets, gpair.Shape(1));
  linalg::Vector<GradientPairPrecise> sum = linalg::Constant(ctx, GradientPairPrecise{}, n_targets);
  CHECK(out.Contiguous());

  // Reduce by column
  auto key_it = dh::MakeTransformIterator<bst_target_t>(
      thrust::make_counting_iterator(0ul),
      [=] XGBOOST_DEVICE(std::size_t i) -> bst_target_t { return i / gpair.Shape(0); });
  auto grad_it = dh::MakeTransformIterator<GradientPairPrecise>(
      thrust::make_counting_iterator(0ul),
      [=] XGBOOST_DEVICE(std::size_t i) -> GradientPairPrecise {
        auto target = i / gpair.Shape(0);
        auto sample = i % gpair.Shape(0);
        return GradientPairPrecise{gpair(sample, target)};
      });
  auto d_sum = sum.View(ctx->gpu_id);
  CHECK(d_sum.CContiguous());

  dh::XGBCachingDeviceAllocator<char> alloc;
  auto policy = thrust::cuda::par(alloc);
  thrust::reduce_by_key(policy, key_it, key_it + gpair.Size(), grad_it,
                        thrust::make_discard_iterator(), dh::tbegin(d_sum.Values()));

  collective::AllReduce<collective::Operation::kSum>(
      ctx->gpu_id, reinterpret_cast<double*>(d_sum.Values().data()), d_sum.Size() * 2);

  thrust::for_each_n(policy, thrust::make_counting_iterator(0ul), n_targets,
                     [=] XGBOOST_DEVICE(std::size_t i) mutable {
                       out(i) = static_cast<float>(
                           CalcUnregularizedWeight(d_sum(i).GetGrad(), d_sum(i).GetHess()));
                     });
}
}  // namespace cuda_impl
}  // namespace tree
}  // namespace xgboost
