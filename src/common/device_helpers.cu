/**
 * Copyright 2024, XGBoost contributors
 */
#include "cuda_rt_utils.h"  // for RtVersion
#include "device_helpers.cuh"
#include "xgboost/windefs.h"  // for xgboost_IS_WIN

namespace dh {
PinnedMemory::PinnedMemory() {
#if defined(xgboost_IS_WIN)
  this->impl_.emplace<detail::GrowOnlyPinnedMemoryImpl>();
#else
  std::int32_t major{0}, minor{0};
  xgboost::curt::DrVersion(&major, &minor);
  // Host NUMA allocation requires driver that supports CTK >= 12.5 to be stable.
  if (major >= 12 && minor >= 5) {
    this->impl_.emplace<detail::GrowOnlyVirtualMemVec>(hipMemLocationTypeHostNuma);
  } else {
    this->impl_.emplace<detail::GrowOnlyPinnedMemoryImpl>();
  }
#endif
}
}  // namespace dh
