/*!
 * Copyright 2017-2019 XGBoost contributors
 *
 * \brief Utilities for CUDA.
 */
#ifdef XGBOOST_USE_NCCL
#include <nccl.h>
#endif  // #ifdef XGBOOST_USE_NCCL
#include <sstream>

#include "device_helpers.cuh"

namespace dh {

#if __CUDACC_VER_MAJOR__ > 9
constexpr std::size_t kUuidLength =
    sizeof(std::declval<hipDeviceProp_t>().uuid) / sizeof(uint64_t);

void GetCudaUUID(int world_size, int rank, int device_ord,
                 xgboost::common::Span<uint64_t, kUuidLength> uuid) {
  hipDeviceProp_t prob;
  safe_cuda(hipGetDeviceProperties(&prob, device_ord));
  std::memcpy(uuid.data(), static_cast<void*>(&(prob.uuid)), sizeof(prob.uuid));
}

std::string PrintUUID(xgboost::common::Span<uint64_t, kUuidLength> uuid) {
  std::stringstream ss;
  for (auto v : uuid) {
    ss << std::hex << v;
  }
  return ss.str();
}

#endif  // __CUDACC_VER_MAJOR__ > 9

void AllReducer::Init(int _device_ordinal) {
#ifdef XGBOOST_USE_NCCL
  LOG(DEBUG) << "Running nccl init on: " << __CUDACC_VER_MAJOR__ << "." << __CUDACC_VER_MINOR__;

  device_ordinal_ = _device_ordinal;
  int32_t const rank = rabit::GetRank();

#if __CUDACC_VER_MAJOR__ > 9
  int32_t const world = rabit::GetWorldSize();

  std::vector<uint64_t> uuids(world * kUuidLength, 0);
  auto s_uuid = xgboost::common::Span<uint64_t>{uuids.data(), uuids.size()};
  auto s_this_uuid = s_uuid.subspan(rank * kUuidLength, kUuidLength);
  GetCudaUUID(world, rank, device_ordinal_, s_this_uuid);

  // No allgather yet.
  rabit::Allreduce<rabit::op::Sum, uint64_t>(uuids.data(), uuids.size());

  std::vector<xgboost::common::Span<uint64_t, kUuidLength>> converted(world);;
  size_t j = 0;
  for (size_t i = 0; i < uuids.size(); i += kUuidLength) {
    converted[j] =
        xgboost::common::Span<uint64_t, kUuidLength>{uuids.data() + i, kUuidLength};
    j++;
  }

  auto iter = std::unique(converted.begin(), converted.end());
  auto n_uniques = std::distance(converted.begin(), iter);
  CHECK_EQ(n_uniques, world)
      << "Multiple processes within communication group running on same CUDA "
      << "device is not supported";
#endif  // __CUDACC_VER_MAJOR__ > 9

  id_ = GetUniqueId();
  dh::safe_cuda(hipSetDevice(device_ordinal_));
  dh::safe_nccl(ncclCommInitRank(&comm_, rabit::GetWorldSize(), id_, rank));
  safe_cuda(hipStreamCreate(&stream_));
  initialised_ = true;
#else
  if (rabit::IsDistributed()) {
    LOG(FATAL) << "XGBoost is not compiled with NCCL.";
  }
#endif  // XGBOOST_USE_NCCL
}

void AllReducer::AllGather(void const *data, size_t length_bytes,
                           std::vector<size_t> *segments,
                           dh::caching_device_vector<char> *recvbuf) {
#ifdef XGBOOST_USE_NCCL
  CHECK(initialised_);
  dh::safe_cuda(hipSetDevice(device_ordinal_));
  size_t world = rabit::GetWorldSize();
  segments->clear();
  segments->resize(world, 0);
  segments->at(rabit::GetRank()) = length_bytes;
  rabit::Allreduce<rabit::op::Max>(segments->data(), segments->size());
  auto total_bytes = std::accumulate(segments->cbegin(), segments->cend(), 0);
  recvbuf->resize(total_bytes);

  size_t offset = 0;
  safe_nccl(ncclGroupStart());
  for (int32_t i = 0; i < world; ++i) {
    size_t as_bytes = segments->at(i);
    safe_nccl(
        ncclBroadcast(data, recvbuf->data().get() + offset,
                      as_bytes, ncclChar, i, comm_, stream_));
    offset += as_bytes;
  }
  safe_nccl(ncclGroupEnd());
#endif  // XGBOOST_USE_NCCL
}

AllReducer::~AllReducer() {
#ifdef XGBOOST_USE_NCCL
  if (initialised_) {
    dh::safe_cuda(hipStreamDestroy(stream_));
    ncclCommDestroy(comm_);
  }
  if (xgboost::ConsoleLogger::ShouldLog(xgboost::ConsoleLogger::LV::kDebug)) {
    LOG(CONSOLE) << "======== NCCL Statistics========";
    LOG(CONSOLE) << "AllReduce calls: " << allreduce_calls_;
    LOG(CONSOLE) << "AllReduce total MiB communicated: " << allreduce_bytes_/1048576;
  }
#endif  // XGBOOST_USE_NCCL
}

}  // namespace dh
