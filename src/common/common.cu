/*!
 * Copyright 2018 XGBoost contributors
 */
#include "common.h"

namespace xgboost {
namespace common {

int AllVisibleGPUs() {
  int n_visgpus = 0;
  try {
    // When compiled with CUDA but running on CPU only device,
    // hipGetDeviceCount will fail.
    dh::safe_cuda(hipGetDeviceCount(&n_visgpus));
  } catch(const dmlc::Error &except) {
    return 0;
  }
  return n_visgpus;
}

}  // namespace common
}  // namespace xgboost
