/*!
 * Copyright 2018-2019 XGBoost contributors
 */
#include "common.h"
#include "xgboost/gpuset.h"

namespace xgboost {

int AllVisibleImpl::AllVisible() {
  int n_visgpus = 0;
  try {
    // When compiled with CUDA but running on CPU only device,
    // hipGetDeviceCount will fail.
    dh::safe_cuda(hipGetDeviceCount(&n_visgpus));
  } catch(const dmlc::Error &except) {
    return 0;
  }
  return n_visgpus;
}

}  // namespace xgboost
