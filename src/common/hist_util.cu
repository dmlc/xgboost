#include "hip/hip_runtime.h"
/*!
 * Copyright 2018 XGBoost contributors
 */

#include "./hist_util.h"

#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>

#include <utility>
#include <vector>

#include "../tree/param.h"
#include "./host_device_vector.h"
#include "./device_helpers.cuh"
#include "./quantile.h"

namespace xgboost {
namespace common {

using WXQSketch = HistCutMatrix::WXQSketch;

__global__ void find_cuts_k
(WXQSketch::Entry* __restrict__ cuts, const bst_float* __restrict__ data,
 const float* __restrict__ cum_weights, int nsamples, int ncuts) {
  // ncuts < nsamples
  int icut = threadIdx.x + blockIdx.x * blockDim.x;
  if (icut >= ncuts)
    return;
  WXQSketch::Entry v;
  int isample = 0;
  if (icut == 0) {
    isample = 0;
  } else if (icut == ncuts - 1) {
    isample = nsamples - 1;
  } else {
    bst_float rank = cum_weights[nsamples - 1] / static_cast<float>(ncuts - 1)
      * static_cast<float>(icut);
    // -1 is used because cum_weights is an inclusive sum
    isample = dh::UpperBound(cum_weights, nsamples, rank);
    isample = max(0, min(isample, nsamples - 1));
  }
  // repeated values will be filtered out on the CPU
  bst_float rmin = isample > 0 ? cum_weights[isample - 1] : 0;
  bst_float rmax = cum_weights[isample];
  cuts[icut] = WXQSketch::Entry(rmin, rmax, rmax - rmin, data[isample]);
}

// predictate for thrust filtering that returns true if the element is not a NaN
struct IsNotNaN {
  __device__ bool operator()(float a) const { return !isnan(a); }
};

__global__ void unpack_features_k
(float* __restrict__ fvalues, float* __restrict__ feature_weights,
 const size_t* __restrict__ row_ptrs, const float* __restrict__ weights,
 Entry* entries, size_t nrows_array, int ncols, size_t row_begin_ptr,
 size_t nrows) {
  size_t irow = threadIdx.x + size_t(blockIdx.x) * blockDim.x;
  if (irow >= nrows) {
    return;
  }
  size_t row_length = row_ptrs[irow + 1] - row_ptrs[irow];
  int icol = threadIdx.y + blockIdx.y * blockDim.y;
  if (icol >= row_length) {
    return;
  }
  Entry entry = entries[row_ptrs[irow] - row_begin_ptr + icol];
  size_t ind = entry.index * nrows_array + irow;
  // if weights are present, ensure that a non-NaN value is written to weights
  // if and only if it is also written to features
  if (!isnan(entry.fvalue) && (weights == nullptr || !isnan(weights[irow]))) {
    fvalues[ind] = entry.fvalue;
    if (feature_weights != nullptr) {
      feature_weights[ind] = weights[irow];
    }
  }
}

// finds quantiles on the GPU
struct GPUSketcher {
  // manage memory for a single GPU
  struct DeviceShard {
    int device_;
    bst_uint row_begin_;  // The row offset for this shard
    bst_uint row_end_;
    bst_uint n_rows_;
    int num_cols_{0};
    size_t n_cuts_{0};
    size_t gpu_batch_nrows_{0};
    bool has_weights_{false};

    tree::TrainParam param_;
    std::vector<WXQSketch> sketches_;
    thrust::device_vector<size_t> row_ptrs_;
    std::vector<WXQSketch::SummaryContainer> summaries_;
    thrust::device_vector<Entry> entries_;
    thrust::device_vector<bst_float> fvalues_;
    thrust::device_vector<bst_float> feature_weights_;
    thrust::device_vector<bst_float> fvalues_cur_;
    thrust::device_vector<WXQSketch::Entry> cuts_d_;
    thrust::host_vector<WXQSketch::Entry> cuts_h_;
    thrust::device_vector<bst_float> weights_;
    thrust::device_vector<bst_float> weights2_;
    std::vector<size_t> n_cuts_cur_;
    thrust::device_vector<size_t> num_elements_;
    thrust::device_vector<char> tmp_storage_;

    DeviceShard(int device, bst_uint row_begin, bst_uint row_end,
                tree::TrainParam param) :
      device_(device), row_begin_(row_begin), row_end_(row_end),
      n_rows_(row_end - row_begin), param_(std::move(param)) {
    }

    void Init(const SparsePage& row_batch, const MetaInfo& info) {
      num_cols_ = info.num_col_;
      has_weights_ = info.weights_.size() > 0;

      // find the batch size
      if (param_.gpu_batch_nrows == 0) {
        // By default, use no more than 1/16th of GPU memory
        gpu_batch_nrows_ = dh::TotalMemory(device_) /
          (16 * num_cols_ * sizeof(Entry));
      } else if (param_.gpu_batch_nrows == -1) {
        gpu_batch_nrows_ = n_rows_;
      } else {
        gpu_batch_nrows_ = param_.gpu_batch_nrows;
      }
      if (gpu_batch_nrows_ > n_rows_) {
        gpu_batch_nrows_ = n_rows_;
      }

      // initialize sketches
      sketches_.resize(num_cols_);
      summaries_.resize(num_cols_);
      constexpr int kFactor = 8;
      double eps = 1.0 / (kFactor * param_.max_bin);
      size_t dummy_nlevel;
      WXQSketch::LimitSizeLevel(row_batch.Size(), eps, &dummy_nlevel, &n_cuts_);
      // double ncuts to be the same as the number of values
      // in the temporary buffers of the sketches
      n_cuts_ *= 2;
      for (int icol = 0; icol < num_cols_; ++icol) {
        sketches_[icol].Init(row_batch.Size(), eps);
        summaries_[icol].Reserve(n_cuts_);
      }

      // allocate necessary GPU buffers
      dh::safe_cuda(hipSetDevice(device_));

      entries_.resize(gpu_batch_nrows_ * num_cols_);
      fvalues_.resize(gpu_batch_nrows_ * num_cols_);
      fvalues_cur_.resize(gpu_batch_nrows_);
      cuts_d_.resize(n_cuts_ * num_cols_);
      cuts_h_.resize(n_cuts_ * num_cols_);
      weights_.resize(gpu_batch_nrows_);
      weights2_.resize(gpu_batch_nrows_);
      num_elements_.resize(1);

      if (has_weights_) {
        feature_weights_.resize(gpu_batch_nrows_ * num_cols_);
      }
      n_cuts_cur_.resize(num_cols_);

      // allocate storage for CUB algorithms; the size is the maximum of the sizes
      // required for various algorithm
      size_t tmp_size = 0, cur_tmp_size = 0;
      // size for sorting
      if (has_weights_) {
        hipcub::DeviceRadixSort::SortPairs
          (nullptr, cur_tmp_size, fvalues_cur_.data().get(),
           fvalues_.data().get(), weights_.data().get(), weights2_.data().get(),
           gpu_batch_nrows_);
      } else {
        hipcub::DeviceRadixSort::SortKeys
          (nullptr, cur_tmp_size, fvalues_cur_.data().get(), fvalues_.data().get(),
           gpu_batch_nrows_);
      }
      tmp_size = std::max(tmp_size, cur_tmp_size);
      // size for inclusive scan
      if (has_weights_) {
        hipcub::DeviceScan::InclusiveSum
          (nullptr, cur_tmp_size, weights2_.begin(), weights_.begin(), gpu_batch_nrows_);
        tmp_size = std::max(tmp_size, cur_tmp_size);
      }
      // size for reduction by key
      hipcub::DeviceReduce::ReduceByKey
        (nullptr, cur_tmp_size, fvalues_.begin(),
         fvalues_cur_.begin(), weights_.begin(), weights2_.begin(),
         num_elements_.begin(), thrust::maximum<bst_float>(), gpu_batch_nrows_);
      tmp_size = std::max(tmp_size, cur_tmp_size);
      // size for filtering
      hipcub::DeviceSelect::If
        (nullptr, cur_tmp_size, fvalues_.begin(), fvalues_cur_.begin(),
         num_elements_.begin(), gpu_batch_nrows_, IsNotNaN());
      tmp_size = std::max(tmp_size, cur_tmp_size);

      tmp_storage_.resize(tmp_size);
    }

    void FindColumnCuts(size_t batch_nrows, size_t icol) {
      size_t tmp_size = tmp_storage_.size();
      // filter out NaNs in feature values
      auto fvalues_begin = fvalues_.data() + icol * gpu_batch_nrows_;
      hipcub::DeviceSelect::If
        (tmp_storage_.data().get(), tmp_size, fvalues_begin,
         fvalues_cur_.data(), num_elements_.begin(), batch_nrows, IsNotNaN());
      size_t nfvalues_cur = 0;
      thrust::copy_n(num_elements_.begin(), 1, &nfvalues_cur);

      // compute cumulative weights using a prefix scan
      if (has_weights_) {
        // filter out NaNs in weights;
        // since hipcub::DeviceSelect::If performs stable filtering,
        // the weights are stored in the correct positions
        auto feature_weights_begin = feature_weights_.data() +
          icol * gpu_batch_nrows_;
        hipcub::DeviceSelect::If
          (tmp_storage_.data().get(), tmp_size, feature_weights_begin,
           weights_.data().get(), num_elements_.begin(), batch_nrows, IsNotNaN());

        // sort the values and weights
        hipcub::DeviceRadixSort::SortPairs
          (tmp_storage_.data().get(), tmp_size, fvalues_cur_.data().get(),
           fvalues_begin.get(), weights_.data().get(), weights2_.data().get(),
           nfvalues_cur);

        // sum the weights to get cumulative weight values
        hipcub::DeviceScan::InclusiveSum
          (tmp_storage_.data().get(), tmp_size, weights2_.begin(),
           weights_.begin(), nfvalues_cur);
      } else {
        // sort the batch values
        hipcub::DeviceRadixSort::SortKeys
          (tmp_storage_.data().get(), tmp_size,
           fvalues_cur_.data().get(), fvalues_begin.get(), nfvalues_cur);

        // fill in cumulative weights with counting iterator
        thrust::copy_n(thrust::make_counting_iterator(1), nfvalues_cur,
                       weights_.begin());
      }

      // remove repeated items and sum the weights across them;
      // non-negative weights are assumed
      hipcub::DeviceReduce::ReduceByKey
        (tmp_storage_.data().get(), tmp_size, fvalues_begin,
         fvalues_cur_.begin(), weights_.begin(), weights2_.begin(),
         num_elements_.begin(), thrust::maximum<bst_float>(), nfvalues_cur);
      size_t n_unique = 0;
      thrust::copy_n(num_elements_.begin(), 1, &n_unique);

      // extract cuts
      n_cuts_cur_[icol] = std::min(n_cuts_, n_unique);
      // if less elements than cuts: copy all elements with their weights
      if (n_cuts_ > n_unique) {
        auto weights2_iter = weights2_.begin();
        auto fvalues_iter = fvalues_cur_.begin();
        auto cuts_iter = cuts_d_.begin() + icol * n_cuts_;
        dh::LaunchN(device_, n_unique, [=]__device__(size_t i) {
            bst_float rmax = weights2_iter[i];
            bst_float rmin = i > 0 ? weights2_iter[i - 1] : 0;
            cuts_iter[i] = WXQSketch::Entry(rmin, rmax, rmax - rmin, fvalues_iter[i]);
          });
      } else if (n_cuts_cur_[icol] > 0) {
        // if more elements than cuts: use binary search on cumulative weights
        int block = 256;
        find_cuts_k<<<dh::DivRoundUp(n_cuts_cur_[icol], block), block>>>
          (cuts_d_.data().get() + icol * n_cuts_, fvalues_cur_.data().get(),
           weights2_.data().get(), n_unique, n_cuts_cur_[icol]);
        dh::safe_cuda(hipGetLastError());
      }
    }

    void SketchBatch(const SparsePage& row_batch, const MetaInfo& info,
                     size_t gpu_batch) {
      // compute start and end indices
      size_t batch_row_begin = gpu_batch * gpu_batch_nrows_;
      size_t batch_row_end = std::min((gpu_batch + 1) * gpu_batch_nrows_,
                                      static_cast<size_t>(n_rows_));
      size_t batch_nrows = batch_row_end - batch_row_begin;
      size_t n_entries =
        row_batch.offset[row_begin_ + batch_row_end] -
        row_batch.offset[row_begin_ + batch_row_begin];
      // copy the batch to the GPU
      dh::safe_cuda
        (hipMemcpy(entries_.data().get(),
                    &row_batch.data[row_batch.offset[row_begin_ + batch_row_begin]],
                    n_entries * sizeof(Entry), hipMemcpyDefault));
      // copy the weights if necessary
      if (has_weights_) {
        dh::safe_cuda
          (hipMemcpy(weights_.data().get(),
                      info.weights_.data() + row_begin_ + batch_row_begin,
                      batch_nrows * sizeof(bst_float), hipMemcpyDefault));
      }

      // unpack the features; also unpack weights if present
      thrust::fill(fvalues_.begin(), fvalues_.end(), NAN);
      thrust::fill(feature_weights_.begin(), feature_weights_.end(), NAN);

      dim3 block3(64, 4, 1);
      dim3 grid3(dh::DivRoundUp(batch_nrows, block3.x),
                 dh::DivRoundUp(num_cols_, block3.y), 1);
      unpack_features_k<<<grid3, block3>>>
        (fvalues_.data().get(), has_weights_ ? feature_weights_.data().get() : nullptr,
         row_ptrs_.data().get() + batch_row_begin,
         has_weights_ ? weights_.data().get() : nullptr, entries_.data().get(),
         gpu_batch_nrows_, num_cols_,
         row_batch.offset[row_begin_ + batch_row_begin], batch_nrows);
      dh::safe_cuda(hipGetLastError());
      dh::safe_cuda(hipDeviceSynchronize());

      for (int icol = 0; icol < num_cols_; ++icol) {
        FindColumnCuts(batch_nrows, icol);
      }

      dh::safe_cuda(hipDeviceSynchronize());

      // add cuts into sketches
      thrust::copy(cuts_d_.begin(), cuts_d_.end(), cuts_h_.begin());
      for (int icol = 0; icol < num_cols_; ++icol) {
        summaries_[icol].MakeFromSorted(&cuts_h_[n_cuts_ * icol], n_cuts_cur_[icol]);
        sketches_[icol].PushSummary(summaries_[icol]);
      }
    }

    void Sketch(const SparsePage& row_batch, const MetaInfo& info) {
      // copy rows to the device
      dh::safe_cuda(hipSetDevice(device_));
      row_ptrs_.resize(n_rows_ + 1);
      thrust::copy(row_batch.offset.data() + row_begin_,
                   row_batch.offset.data() + row_end_ + 1,
                   row_ptrs_.begin());

      size_t gpu_nbatches = dh::DivRoundUp(n_rows_, gpu_batch_nrows_);

      for (size_t gpu_batch = 0; gpu_batch < gpu_nbatches; ++gpu_batch) {
        SketchBatch(row_batch, info, gpu_batch);
      }
    }
  };

  void Sketch(const SparsePage& batch, const MetaInfo& info, HistCutMatrix* hmat) {
    // partition input matrix into row segments
    std::vector<size_t> row_segments;
    dh::RowSegments(info.num_row_, devices_.Size(), &row_segments);

    // create device shards
    shards_.resize(devices_.Size());
    dh::ExecuteIndexShards(&shards_, [&](int i, std::unique_ptr<DeviceShard>& shard) {
        shard = std::unique_ptr<DeviceShard>
          (new DeviceShard(devices_[i], row_segments[i], row_segments[i + 1], param_));
      });

    // compute sketches for each shard
    dh::ExecuteShards(&shards_, [&](std::unique_ptr<DeviceShard>& shard) {
        shard->Init(batch, info);
        shard->Sketch(batch, info);
      });

    // merge the sketches from all shards
    // TODO(canonizer): do it in a tree-like reduction
    int num_cols = info.num_col_;
    std::vector<WXQSketch> sketches(num_cols);
    WXQSketch::SummaryContainer summary;
    for (int icol = 0; icol < num_cols; ++icol) {
      sketches[icol].Init(batch.Size(), 1.0 / (8 * param_.max_bin));
      for (int shard = 0; shard < shards_.size(); ++shard) {
        shards_[shard]->sketches_[icol].GetSummary(&summary);
        sketches[icol].PushSummary(summary);
      }
    }

    hmat->Init(&sketches, param_.max_bin);
  }

  GPUSketcher(tree::TrainParam param, size_t n_rows) : param_(std::move(param)) {
    devices_ = GPUSet::Range(param_.gpu_id, dh::NDevices(param_.n_gpus, n_rows));
  }

  std::vector<std::unique_ptr<DeviceShard>> shards_;
  tree::TrainParam param_;
  GPUSet devices_;
};

void DeviceSketch
  (const SparsePage& batch, const MetaInfo& info,
   const tree::TrainParam& param, HistCutMatrix* hmat) {
  GPUSketcher sketcher(param, info.num_row_);
  sketcher.Sketch(batch, info, hmat);
}

}  // namespace common
}  // namespace xgboost
