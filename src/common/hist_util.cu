#include "hip/hip_runtime.h"
/*!
 * Copyright 2018~2020 XGBoost contributors
 */

#include <xgboost/logging.h>

#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

#include <memory>
#include <mutex>
#include <utility>
#include <vector>

#include "device_helpers.cuh"
#include "hist_util.h"
#include "hist_util.cuh"
#include "math.h"  // NOLINT
#include "quantile.h"
#include "categorical.h"
#include "xgboost/host_device_vector.h"


namespace xgboost {
namespace common {

constexpr float SketchContainer::kFactor;

namespace detail {
size_t RequiredSampleCutsPerColumn(int max_bins, size_t num_rows) {
  double eps = 1.0 / (WQSketch::kFactor * max_bins);
  size_t dummy_nlevel;
  size_t num_cuts;
  WQuantileSketch<bst_float, bst_float>::LimitSizeLevel(
      num_rows, eps, &dummy_nlevel, &num_cuts);
  return std::min(num_cuts, num_rows);
}

size_t RequiredSampleCuts(bst_row_t num_rows, bst_feature_t num_columns,
                          size_t max_bins, size_t nnz) {
  auto per_column = RequiredSampleCutsPerColumn(max_bins, num_rows);
  auto if_dense = num_columns * per_column;
  auto result = std::min(nnz, if_dense);
  return result;
}

size_t RequiredMemory(bst_row_t num_rows, bst_feature_t num_columns, size_t nnz,
                      size_t num_bins, bool with_weights) {
  size_t peak = 0;
  // 0. Allocate cut pointer in quantile container by increasing: n_columns + 1
  size_t total = (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 1. Copy and sort: 2 * bytes_per_element * shape
  total += BytesPerElement(with_weights) * num_rows * num_columns;
  peak = std::max(peak, total);
  // 2. Deallocate bytes_per_element * shape due to reusing memory in sort.
  total -= BytesPerElement(with_weights) * num_rows * num_columns / 2;
  // 3. Allocate colomn size scan by increasing: n_columns + 1
  total += (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 4. Allocate cut pointer by increasing: n_columns + 1
  total += (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 5. Allocate cuts: assuming rows is greater than bins: n_columns * limit_size
  total += RequiredSampleCuts(num_rows, num_bins, num_bins, nnz) * sizeof(SketchEntry);
  // 6. Deallocate copied entries by reducing: bytes_per_element * shape.
  peak = std::max(peak, total);
  total -= (BytesPerElement(with_weights) * num_rows * num_columns) / 2;
  // 7. Deallocate column size scan.
  peak = std::max(peak, total);
  total -= (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 8. Deallocate cut size scan.
  total -= (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 9. Allocate final cut values, min values, cut ptrs: std::min(rows, bins + 1) *
  //    n_columns + n_columns + n_columns + 1
  total += std::min(num_rows, num_bins) * num_columns * sizeof(float);
  total += num_columns *
           sizeof(std::remove_reference_t<decltype(
                      std::declval<HistogramCuts>().MinValues())>::value_type);
  total += (num_columns + 1) *
           sizeof(std::remove_reference_t<decltype(
                      std::declval<HistogramCuts>().Ptrs())>::value_type);
  peak = std::max(peak, total);

  return peak;
}

size_t SketchBatchNumElements(size_t sketch_batch_num_elements,
                              bst_row_t num_rows, bst_feature_t columns,
                              size_t nnz, int device,
                              size_t num_cuts, bool has_weight) {
#if defined(XGBOOST_USE_RMM) && XGBOOST_USE_RMM == 1
  // device available memory is not accurate when rmm is used.
  return nnz;
#endif  // defined(XGBOOST_USE_RMM) && XGBOOST_USE_RMM == 1

  if (sketch_batch_num_elements == 0) {
    auto required_memory = RequiredMemory(num_rows, columns, nnz, num_cuts, has_weight);
    // use up to 80% of available space
    auto avail = dh::AvailableMemory(device) * 0.8;
    if (required_memory > avail) {
      sketch_batch_num_elements = avail / BytesPerElement(has_weight);
    } else {
      sketch_batch_num_elements = std::min(num_rows * static_cast<size_t>(columns), nnz);
    }
  }
  return sketch_batch_num_elements;
}

void SortByWeight(dh::device_vector<float>* weights,
                  dh::device_vector<Entry>* sorted_entries) {
  // Sort both entries and wegihts.
  dh::XGBDeviceAllocator<char> alloc;
  thrust::sort_by_key(thrust::cuda::par(alloc), sorted_entries->begin(),
                      sorted_entries->end(), weights->begin(),
                      detail::EntryCompareOp());

  // Scan weights
  dh::XGBCachingDeviceAllocator<char> caching;
  thrust::inclusive_scan_by_key(thrust::cuda::par(caching),
                                sorted_entries->begin(), sorted_entries->end(),
                                weights->begin(), weights->begin(),
                                [=] __device__(const Entry& a, const Entry& b) {
                                  return a.index == b.index;
                                });
}

void RemoveDuplicatedCategories(
    int32_t device, MetaInfo const &info, Span<bst_row_t> d_cuts_ptr,
    dh::device_vector<Entry> *p_sorted_entries,
    dh::caching_device_vector<size_t> *p_column_sizes_scan) {
  info.feature_types.SetDevice(device);
  auto d_feature_types = info.feature_types.ConstDeviceSpan();
  CHECK(!d_feature_types.empty());
  auto &column_sizes_scan = *p_column_sizes_scan;
  auto &sorted_entries = *p_sorted_entries;
  // Removing duplicated entries in categorical features.
  dh::caching_device_vector<size_t> new_column_scan(column_sizes_scan.size());
  dh::SegmentedUnique(column_sizes_scan.data().get(),
                      column_sizes_scan.data().get() + column_sizes_scan.size(),
                      sorted_entries.begin(), sorted_entries.end(),
                      new_column_scan.data().get(), sorted_entries.begin(),
                      [=] __device__(Entry const &l, Entry const &r) {
                        if (l.index == r.index) {
                          if (IsCat(d_feature_types, l.index)) {
                            return l.fvalue == r.fvalue;
                          }
                        }
                        return false;
                      });

  // Renew the column scan and cut scan based on categorical data.
  auto d_old_column_sizes_scan = dh::ToSpan(column_sizes_scan);
  dh::caching_device_vector<SketchContainer::OffsetT> new_cuts_size(
      info.num_col_ + 1);
  CHECK_EQ(new_column_scan.size(), new_cuts_size.size());
  dh::LaunchN(
      new_column_scan.size(),
      [=, d_new_cuts_size = dh::ToSpan(new_cuts_size),
       d_old_column_sizes_scan = dh::ToSpan(column_sizes_scan),
       d_new_columns_ptr = dh::ToSpan(new_column_scan)] __device__(size_t idx) {
        d_old_column_sizes_scan[idx] = d_new_columns_ptr[idx];
        if (idx == d_new_columns_ptr.size() - 1) {
          return;
        }
        if (IsCat(d_feature_types, idx)) {
          // Cut size is the same as number of categories in input.
          d_new_cuts_size[idx] =
              d_new_columns_ptr[idx + 1] - d_new_columns_ptr[idx];
        } else {
          d_new_cuts_size[idx] = d_cuts_ptr[idx + 1] - d_cuts_ptr[idx];
        }
      });
  // Turn size into ptr.
  thrust::exclusive_scan(thrust::device, new_cuts_size.cbegin(),
                         new_cuts_size.cend(), d_cuts_ptr.data());
}
}  // namespace detail

void ProcessBatch(int device, MetaInfo const &info, const SparsePage &page,
                  size_t begin, size_t end, SketchContainer *sketch_container,
                  int num_cuts_per_feature, size_t num_columns) {
  dh::XGBCachingDeviceAllocator<char> alloc;
  dh::device_vector<Entry> sorted_entries;
  if (page.data.DeviceCanRead()) {
    const auto& device_data = page.data.ConstDevicePointer();
    sorted_entries = dh::device_vector<Entry>(device_data + begin, device_data + end);
  } else {
    const auto& host_data = page.data.ConstHostVector();
    sorted_entries = dh::device_vector<Entry>(host_data.begin() + begin,
                                              host_data.begin() + end);
  }
  thrust::sort(thrust::cuda::par(alloc), sorted_entries.begin(),
               sorted_entries.end(), detail::EntryCompareOp());

  HostDeviceVector<SketchContainer::OffsetT> cuts_ptr;
  dh::caching_device_vector<size_t> column_sizes_scan;
  data::IsValidFunctor dummy_is_valid(std::numeric_limits<float>::quiet_NaN());
  auto batch_it = dh::MakeTransformIterator<data::COOTuple>(
      sorted_entries.data().get(),
      [] __device__(Entry const &e) -> data::COOTuple {
        return {0, e.index, e.fvalue};  // row_idx is not needed for scanning column size.
      });
  detail::GetColumnSizesScan(device, num_columns, num_cuts_per_feature,
                             batch_it, dummy_is_valid,
                             0, sorted_entries.size(),
                             &cuts_ptr, &column_sizes_scan);
  auto d_cuts_ptr = cuts_ptr.DeviceSpan();

  if (sketch_container->HasCategorical()) {
    detail::RemoveDuplicatedCategories(device, info, d_cuts_ptr,
                                       &sorted_entries, &column_sizes_scan);
  }

  auto const& h_cuts_ptr = cuts_ptr.ConstHostVector();
  CHECK_EQ(d_cuts_ptr.size(), column_sizes_scan.size());

  // add cuts into sketches
  sketch_container->Push(dh::ToSpan(sorted_entries), dh::ToSpan(column_sizes_scan),
                         d_cuts_ptr, h_cuts_ptr.back());
  sorted_entries.clear();
  sorted_entries.shrink_to_fit();
  CHECK_EQ(sorted_entries.capacity(), 0);
  CHECK_NE(cuts_ptr.Size(), 0);
}

void ProcessWeightedBatch(int device, const SparsePage& page,
                          MetaInfo const& info, size_t begin, size_t end,
                          SketchContainer* sketch_container, int num_cuts_per_feature,
                          size_t num_columns,
                          bool is_ranking, Span<bst_group_t const> d_group_ptr) {
  auto weights = info.weights_.ConstDeviceSpan();

  dh::XGBCachingDeviceAllocator<char> alloc;
  const auto& host_data = page.data.ConstHostVector();
  dh::device_vector<Entry> sorted_entries(host_data.begin() + begin,
                                          host_data.begin() + end);

  // Binary search to assign weights to each element
  dh::device_vector<float> temp_weights(sorted_entries.size());
  auto d_temp_weights = temp_weights.data().get();
  page.offset.SetDevice(device);
  auto row_ptrs = page.offset.ConstDeviceSpan();
  size_t base_rowid = page.base_rowid;
  if (is_ranking) {
    CHECK_GE(d_group_ptr.size(), 2)
        << "Must have at least 1 group for ranking.";
    CHECK_EQ(weights.size(), d_group_ptr.size() - 1)
        << "Weight size should equal to number of groups.";
    dh::LaunchN(temp_weights.size(), [=] __device__(size_t idx) {
        size_t element_idx = idx + begin;
        size_t ridx = dh::SegmentId(row_ptrs, element_idx);
        bst_group_t group_idx = dh::SegmentId(d_group_ptr, ridx + base_rowid);
        d_temp_weights[idx] = weights[group_idx];
      });
  } else {
    dh::LaunchN(temp_weights.size(), [=] __device__(size_t idx) {
        size_t element_idx = idx + begin;
        size_t ridx = dh::SegmentId(row_ptrs, element_idx);
        d_temp_weights[idx] = weights[ridx + base_rowid];
      });
  }
  detail::SortByWeight(&temp_weights, &sorted_entries);

  HostDeviceVector<SketchContainer::OffsetT> cuts_ptr;
  dh::caching_device_vector<size_t> column_sizes_scan;
  data::IsValidFunctor dummy_is_valid(std::numeric_limits<float>::quiet_NaN());
  auto batch_it = dh::MakeTransformIterator<data::COOTuple>(
      sorted_entries.data().get(),
      [] __device__(Entry const &e) -> data::COOTuple {
        return {0, e.index, e.fvalue};  // row_idx is not needed for scaning column size.
      });
  detail::GetColumnSizesScan(device, num_columns, num_cuts_per_feature,
                             batch_it, dummy_is_valid,
                             0, sorted_entries.size(),
                             &cuts_ptr, &column_sizes_scan);
  auto d_cuts_ptr = cuts_ptr.DeviceSpan();
  if (sketch_container->HasCategorical()) {
    detail::RemoveDuplicatedCategories(device, info, d_cuts_ptr,
                                       &sorted_entries, &column_sizes_scan);
  }

  auto const& h_cuts_ptr = cuts_ptr.ConstHostVector();

  // Extract cuts
  sketch_container->Push(dh::ToSpan(sorted_entries),
                         dh::ToSpan(column_sizes_scan), d_cuts_ptr,
                         h_cuts_ptr.back(), dh::ToSpan(temp_weights));
  sorted_entries.clear();
  sorted_entries.shrink_to_fit();
}

HistogramCuts DeviceSketch(int device, DMatrix* dmat, int max_bins,
                           size_t sketch_batch_num_elements) {
  dmat->Info().feature_types.SetDevice(device);
  dmat->Info().feature_types.ConstDevicePointer();  // pull to device early
  // Configure batch size based on available memory
  bool has_weights = dmat->Info().weights_.Size() > 0;
  size_t num_cuts_per_feature =
      detail::RequiredSampleCutsPerColumn(max_bins, dmat->Info().num_row_);
  sketch_batch_num_elements = detail::SketchBatchNumElements(
      sketch_batch_num_elements,
      dmat->Info().num_row_,
      dmat->Info().num_col_,
      dmat->Info().num_nonzero_,
      device, num_cuts_per_feature, has_weights);

  HistogramCuts cuts;
  SketchContainer sketch_container(dmat->Info().feature_types, max_bins, dmat->Info().num_col_,
                                   dmat->Info().num_row_, device);

  dmat->Info().weights_.SetDevice(device);
  for (const auto& batch : dmat->GetBatches<SparsePage>()) {
    size_t batch_nnz = batch.data.Size();
    auto const& info = dmat->Info();
    for (auto begin = 0ull; begin < batch_nnz; begin += sketch_batch_num_elements) {
      size_t end = std::min(batch_nnz, size_t(begin + sketch_batch_num_elements));
      if (has_weights) {
        bool is_ranking = HostSketchContainer::UseGroup(dmat->Info());
        dh::caching_device_vector<uint32_t> groups(info.group_ptr_.cbegin(),
                                                   info.group_ptr_.cend());
        ProcessWeightedBatch(
            device, batch, dmat->Info(), begin, end,
            &sketch_container,
            num_cuts_per_feature,
            dmat->Info().num_col_,
            is_ranking, dh::ToSpan(groups));
      } else {
        ProcessBatch(device, dmat->Info(), batch, begin, end, &sketch_container,
                     num_cuts_per_feature, dmat->Info().num_col_);
      }
    }
  }
  sketch_container.MakeCuts(&cuts);
  return cuts;
}
}  // namespace common
}  // namespace xgboost
