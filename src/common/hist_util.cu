#include "hip/hip_runtime.h"
/*!
 * Copyright 2018 XGBoost contributors
 */

#include "./hist_util.h"

#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>

#include <utility>
#include <vector>
#include <memory>
#include <mutex>

#include "../tree/param.h"
#include "./host_device_vector.h"
#include "./device_helpers.cuh"
#include "./quantile.h"

namespace xgboost {
namespace common {

using WXQSketch = HistCutMatrix::WXQSketch;

__global__ void FindCutsK
(WXQSketch::Entry* __restrict__ cuts, const bst_float* __restrict__ data,
 const float* __restrict__ cum_weights, int nsamples, int ncuts) {
  // ncuts < nsamples
  int icut = threadIdx.x + blockIdx.x * blockDim.x;
  if (icut >= ncuts) {
    return;
  }
  WXQSketch::Entry v;
  int isample = 0;
  if (icut == 0) {
    isample = 0;
  } else if (icut == ncuts - 1) {
    isample = nsamples - 1;
  } else {
    bst_float rank = cum_weights[nsamples - 1] / static_cast<float>(ncuts - 1)
      * static_cast<float>(icut);
    // -1 is used because cum_weights is an inclusive sum
    isample = dh::UpperBound(cum_weights, nsamples, rank);
    isample = max(0, min(isample, nsamples - 1));
  }
  // repeated values will be filtered out on the CPU
  bst_float rmin = isample > 0 ? cum_weights[isample - 1] : 0;
  bst_float rmax = cum_weights[isample];
  cuts[icut] = WXQSketch::Entry(rmin, rmax, rmax - rmin, data[isample]);
}

// predictate for thrust filtering that returns true if the element is not a NaN
struct IsNotNaN {
  __device__ bool operator()(float a) const { return !isnan(a); }
};

__global__ void UnpackFeaturesK
(float* __restrict__ fvalues, float* __restrict__ feature_weights,
 const size_t* __restrict__ row_ptrs, const float* __restrict__ weights,
 Entry* entries, size_t nrows_array, int ncols, size_t row_begin_ptr,
 size_t nrows) {
  size_t irow = threadIdx.x + size_t(blockIdx.x) * blockDim.x;
  if (irow >= nrows) {
    return;
  }
  size_t row_length = row_ptrs[irow + 1] - row_ptrs[irow];
  int icol = threadIdx.y + blockIdx.y * blockDim.y;
  if (icol >= row_length) {
    return;
  }
  Entry entry = entries[row_ptrs[irow] - row_begin_ptr + icol];
  size_t ind = entry.index * nrows_array + irow;
  // if weights are present, ensure that a non-NaN value is written to weights
  // if and only if it is also written to features
  if (!isnan(entry.fvalue) && (weights == nullptr || !isnan(weights[irow]))) {
    fvalues[ind] = entry.fvalue;
    if (feature_weights != nullptr && weights != nullptr) {
      feature_weights[ind] = weights[irow];
    }
  }
}

/*!
 * \brief A container that holds the device sketches across all
 *  sparse page batches which are distributed to different devices.
 *  As sketches are aggregated by column, the mutex guards
 *  multiple devices pushing sketch summary for the same column
 *  across distinct rows.
 */
struct SketchContainer {
  std::vector<HistCutMatrix::WXQSketch> sketches_;  // NOLINT
  std::vector<std::mutex> col_locks_; // NOLINT
  static constexpr int kOmpNumColsParallelizeLimit = 1000;

  SketchContainer(const tree::TrainParam &param, DMatrix *dmat) :
    col_locks_(dmat->Info().num_col_) {
    const MetaInfo &info = dmat->Info();
    // Initialize Sketches for this dmatrix
    sketches_.resize(info.num_col_);
#pragma omp parallel for schedule(static) if (info.num_col_ > kOmpNumColsParallelizeLimit)
    for (int icol = 0; icol < info.num_col_; ++icol) {
      sketches_[icol].Init(info.num_row_, 1.0 / (8 * param.max_bin));
    }
  }

  // Prevent copying/assigning/moving this as its internals can't be assigned/copied/moved
  SketchContainer(const SketchContainer &) = delete;
  SketchContainer(const SketchContainer &&) = delete;
  SketchContainer &operator=(const SketchContainer &) = delete;
  SketchContainer &operator=(const SketchContainer &&) = delete;
};

// finds quantiles on the GPU
struct GPUSketcher {
  // manage memory for a single GPU
  class DeviceShard {
    int device_;
    bst_uint row_begin_;  // The row offset for this shard
    bst_uint row_end_;
    bst_uint n_rows_;
    int num_cols_{0};
    size_t n_cuts_{0};
    size_t gpu_batch_nrows_{0};
    bool has_weights_{false};
    size_t row_stride_{0};

    tree::TrainParam param_;
    SketchContainer *sketch_container_;
    thrust::device_vector<size_t> row_ptrs_;
    thrust::device_vector<Entry> entries_;
    thrust::device_vector<bst_float> fvalues_;
    thrust::device_vector<bst_float> feature_weights_;
    thrust::device_vector<bst_float> fvalues_cur_;
    thrust::device_vector<WXQSketch::Entry> cuts_d_;
    thrust::host_vector<WXQSketch::Entry> cuts_h_;
    thrust::device_vector<bst_float> weights_;
    thrust::device_vector<bst_float> weights2_;
    std::vector<size_t> n_cuts_cur_;
    thrust::device_vector<size_t> num_elements_;
    thrust::device_vector<char> tmp_storage_;

   public:
    DeviceShard(int device, bst_uint row_begin, bst_uint row_end,
                tree::TrainParam param, SketchContainer *sketch_container) :
      device_(device), row_begin_(row_begin), row_end_(row_end),
      n_rows_(row_end - row_begin), param_(std::move(param)), sketch_container_(sketch_container) {
    }

    inline size_t GetRowStride() const {
       return row_stride_;
    }

    void Init(const SparsePage& row_batch, const MetaInfo& info, int gpu_batch_nrows) {
      num_cols_ = info.num_col_;
      has_weights_ = info.weights_.Size() > 0;

      // find the batch size
      if (gpu_batch_nrows == 0) {
        // By default, use no more than 1/16th of GPU memory
        gpu_batch_nrows_ = dh::TotalMemory(device_) /
          (16 * num_cols_ * sizeof(Entry));
      } else if (gpu_batch_nrows == -1) {
        gpu_batch_nrows_ = n_rows_;
      } else {
        gpu_batch_nrows_ = gpu_batch_nrows;
      }
      if (gpu_batch_nrows_ > n_rows_) {
        gpu_batch_nrows_ = n_rows_;
      }

      constexpr int kFactor = 8;
      double eps = 1.0 / (kFactor * param_.max_bin);
      size_t dummy_nlevel;
      WXQSketch::LimitSizeLevel(gpu_batch_nrows_, eps, &dummy_nlevel, &n_cuts_);

      // allocate necessary GPU buffers
      dh::safe_cuda(hipSetDevice(device_));

      entries_.resize(gpu_batch_nrows_ * num_cols_);
      fvalues_.resize(gpu_batch_nrows_ * num_cols_);
      fvalues_cur_.resize(gpu_batch_nrows_);
      cuts_d_.resize(n_cuts_ * num_cols_);
      cuts_h_.resize(n_cuts_ * num_cols_);
      weights_.resize(gpu_batch_nrows_);
      weights2_.resize(gpu_batch_nrows_);
      num_elements_.resize(1);

      if (has_weights_) {
        feature_weights_.resize(gpu_batch_nrows_ * num_cols_);
      }
      n_cuts_cur_.resize(num_cols_);

      // allocate storage for CUB algorithms; the size is the maximum of the sizes
      // required for various algorithm
      size_t tmp_size = 0, cur_tmp_size = 0;
      // size for sorting
      if (has_weights_) {
        hipcub::DeviceRadixSort::SortPairs
          (nullptr, cur_tmp_size, fvalues_cur_.data().get(),
           fvalues_.data().get(), weights_.data().get(), weights2_.data().get(),
           gpu_batch_nrows_);
      } else {
        hipcub::DeviceRadixSort::SortKeys
          (nullptr, cur_tmp_size, fvalues_cur_.data().get(), fvalues_.data().get(),
           gpu_batch_nrows_);
      }
      tmp_size = std::max(tmp_size, cur_tmp_size);
      // size for inclusive scan
      if (has_weights_) {
        hipcub::DeviceScan::InclusiveSum
          (nullptr, cur_tmp_size, weights2_.begin(), weights_.begin(), gpu_batch_nrows_);
        tmp_size = std::max(tmp_size, cur_tmp_size);
      }
      // size for reduction by key
      hipcub::DeviceReduce::ReduceByKey
        (nullptr, cur_tmp_size, fvalues_.begin(),
         fvalues_cur_.begin(), weights_.begin(), weights2_.begin(),
         num_elements_.begin(), thrust::maximum<bst_float>(), gpu_batch_nrows_);
      tmp_size = std::max(tmp_size, cur_tmp_size);
      // size for filtering
      hipcub::DeviceSelect::If
        (nullptr, cur_tmp_size, fvalues_.begin(), fvalues_cur_.begin(),
         num_elements_.begin(), gpu_batch_nrows_, IsNotNaN());
      tmp_size = std::max(tmp_size, cur_tmp_size);

      tmp_storage_.resize(tmp_size);
    }

    void FindColumnCuts(size_t batch_nrows, size_t icol) {
      size_t tmp_size = tmp_storage_.size();
      // filter out NaNs in feature values
      auto fvalues_begin = fvalues_.data() + icol * gpu_batch_nrows_;
      hipcub::DeviceSelect::If
        (tmp_storage_.data().get(), tmp_size, fvalues_begin,
         fvalues_cur_.data(), num_elements_.begin(), batch_nrows, IsNotNaN());
      size_t nfvalues_cur = 0;
      thrust::copy_n(num_elements_.begin(), 1, &nfvalues_cur);

      // compute cumulative weights using a prefix scan
      if (has_weights_) {
        // filter out NaNs in weights;
        // since hipcub::DeviceSelect::If performs stable filtering,
        // the weights are stored in the correct positions
        auto feature_weights_begin = feature_weights_.data() +
          icol * gpu_batch_nrows_;
        hipcub::DeviceSelect::If
          (tmp_storage_.data().get(), tmp_size, feature_weights_begin,
           weights_.data().get(), num_elements_.begin(), batch_nrows, IsNotNaN());

        // sort the values and weights
        hipcub::DeviceRadixSort::SortPairs
          (tmp_storage_.data().get(), tmp_size, fvalues_cur_.data().get(),
           fvalues_begin.get(), weights_.data().get(), weights2_.data().get(),
           nfvalues_cur);

        // sum the weights to get cumulative weight values
        hipcub::DeviceScan::InclusiveSum
          (tmp_storage_.data().get(), tmp_size, weights2_.begin(),
           weights_.begin(), nfvalues_cur);
      } else {
        // sort the batch values
        hipcub::DeviceRadixSort::SortKeys
          (tmp_storage_.data().get(), tmp_size,
           fvalues_cur_.data().get(), fvalues_begin.get(), nfvalues_cur);

        // fill in cumulative weights with counting iterator
        thrust::copy_n(thrust::make_counting_iterator(1), nfvalues_cur,
                       weights_.begin());
      }

      // remove repeated items and sum the weights across them;
      // non-negative weights are assumed
      hipcub::DeviceReduce::ReduceByKey
        (tmp_storage_.data().get(), tmp_size, fvalues_begin,
         fvalues_cur_.begin(), weights_.begin(), weights2_.begin(),
         num_elements_.begin(), thrust::maximum<bst_float>(), nfvalues_cur);
      size_t n_unique = 0;
      thrust::copy_n(num_elements_.begin(), 1, &n_unique);

      // extract cuts
      n_cuts_cur_[icol] = std::min(n_cuts_, n_unique);
      // if less elements than cuts: copy all elements with their weights
      if (n_cuts_ > n_unique) {
        float* weights2_ptr = weights2_.data().get();
        float* fvalues_ptr = fvalues_cur_.data().get();
        WXQSketch::Entry* cuts_ptr = cuts_d_.data().get() + icol * n_cuts_;
        dh::LaunchN(device_, n_unique, [=]__device__(size_t i) {
            bst_float rmax = weights2_ptr[i];
            bst_float rmin = i > 0 ? weights2_ptr[i - 1] : 0;
            cuts_ptr[i] = WXQSketch::Entry(rmin, rmax, rmax - rmin, fvalues_ptr[i]);
          });
      } else if (n_cuts_cur_[icol] > 0) {
        // if more elements than cuts: use binary search on cumulative weights
        int block = 256;
        FindCutsK<<<dh::DivRoundUp(n_cuts_cur_[icol], block), block>>>
          (cuts_d_.data().get() + icol * n_cuts_, fvalues_cur_.data().get(),
           weights2_.data().get(), n_unique, n_cuts_cur_[icol]);
        dh::safe_cuda(hipGetLastError());  // NOLINT
      }
    }

    void SketchBatch(const SparsePage& row_batch, const MetaInfo& info,
                     size_t gpu_batch) {
      // compute start and end indices
      size_t batch_row_begin = gpu_batch * gpu_batch_nrows_;
      size_t batch_row_end = std::min((gpu_batch + 1) * gpu_batch_nrows_,
                                      static_cast<size_t>(n_rows_));
      size_t batch_nrows = batch_row_end - batch_row_begin;

      const auto& offset_vec = row_batch.offset.HostVector();
      const auto& data_vec = row_batch.data.HostVector();

      size_t n_entries = offset_vec[row_begin_ + batch_row_end] -
        offset_vec[row_begin_ + batch_row_begin];
      // copy the batch to the GPU
      dh::safe_cuda
        (hipMemcpyAsync(entries_.data().get(),
                    data_vec.data() + offset_vec[row_begin_ + batch_row_begin],
                    n_entries * sizeof(Entry), hipMemcpyDefault));
      // copy the weights if necessary
      if (has_weights_) {
        const auto& weights_vec = info.weights_.HostVector();
        dh::safe_cuda
          (hipMemcpyAsync(weights_.data().get(),
                      weights_vec.data() + row_begin_ + batch_row_begin,
                      batch_nrows * sizeof(bst_float), hipMemcpyDefault));
      }

      // unpack the features; also unpack weights if present
      thrust::fill(fvalues_.begin(), fvalues_.end(), NAN);
      if (has_weights_) {
        thrust::fill(feature_weights_.begin(), feature_weights_.end(), NAN);
      }

      dim3 block3(16, 64, 1);
      // NOTE: This will typically support ~ 4M features - 64K*64
      dim3 grid3(dh::DivRoundUp(batch_nrows, block3.x),
                 dh::DivRoundUp(num_cols_, block3.y), 1);
      UnpackFeaturesK<<<grid3, block3>>>
        (fvalues_.data().get(), has_weights_ ? feature_weights_.data().get() : nullptr,
         row_ptrs_.data().get() + batch_row_begin,
         has_weights_ ? weights_.data().get() : nullptr, entries_.data().get(),
         gpu_batch_nrows_, num_cols_,
         offset_vec[row_begin_ + batch_row_begin], batch_nrows);

      for (int icol = 0; icol < num_cols_; ++icol) {
        FindColumnCuts(batch_nrows, icol);
      }

      // add cuts into sketches
      thrust::copy(cuts_d_.begin(), cuts_d_.end(), cuts_h_.begin());
#pragma omp parallel for schedule(static) \
      if (num_cols_ > SketchContainer::kOmpNumColsParallelizeLimit) // NOLINT
      for (int icol = 0; icol < num_cols_; ++icol) {
        WXQSketch::SummaryContainer summary;
        summary.Reserve(n_cuts_);
        summary.MakeFromSorted(&cuts_h_[n_cuts_ * icol], n_cuts_cur_[icol]);

        std::lock_guard<std::mutex> lock(sketch_container_->col_locks_[icol]);
        sketch_container_->sketches_[icol].PushSummary(summary);
      }
    }

    void ComputeRowStride() {
      // Find the row stride for this batch
      auto row_iter = row_ptrs_.begin();
      // Functor for finding the maximum row size for this batch
      auto get_size = [=] __device__(size_t row) {
        return row_iter[row + 1] - row_iter[row];
      }; // NOLINT

      auto counting = thrust::make_counting_iterator(size_t(0));
      using TransformT = thrust::transform_iterator<decltype(get_size),
                                                    decltype(counting), size_t>;
      TransformT row_size_iter = TransformT(counting, get_size);
      row_stride_ = thrust::reduce(row_size_iter, row_size_iter + n_rows_, 0,
                                   thrust::maximum<size_t>());
    }

    void Sketch(const SparsePage& row_batch, const MetaInfo& info) {
      // copy rows to the device
      dh::safe_cuda(hipSetDevice(device_));
      const auto& offset_vec = row_batch.offset.HostVector();
      row_ptrs_.resize(n_rows_ + 1);
      thrust::copy(offset_vec.data() + row_begin_,
                   offset_vec.data() + row_end_ + 1, row_ptrs_.begin());
      size_t gpu_nbatches = dh::DivRoundUp(n_rows_, gpu_batch_nrows_);
      for (size_t gpu_batch = 0; gpu_batch < gpu_nbatches; ++gpu_batch) {
        SketchBatch(row_batch, info, gpu_batch);
      }
    }
  };

  void SketchBatch(const SparsePage &batch, const MetaInfo &info) {
    GPUDistribution dist =
      GPUDistribution::Block(GPUSet::All(learner_param_.gpu_id, learner_param_.n_gpus,
                                         batch.Size()));

    // create device shards
    shards_.resize(dist.Devices().Size());
    dh::ExecuteIndexShards(&shards_, [&](int i, std::unique_ptr<DeviceShard>& shard) {
        size_t start = dist.ShardStart(batch.Size(), i);
        size_t size = dist.ShardSize(batch.Size(), i);
        shard = std::unique_ptr<DeviceShard>(
            new DeviceShard(dist.Devices().DeviceId(i), start,
                            start + size, param_, sketch_container_.get()));
      });

    // compute sketches for each shard
    dh::ExecuteIndexShards(&shards_,
                           [&](int idx, std::unique_ptr<DeviceShard>& shard) {
                             shard->Init(batch, info, gpu_batch_nrows_);
                             shard->Sketch(batch, info);
                             shard->ComputeRowStride();
                           });

    // compute row stride across all shards
    for (const auto &shard : shards_) {
      row_stride_ = std::max(row_stride_, shard->GetRowStride());
    }
  }

  GPUSketcher(const tree::TrainParam &param, const LearnerTrainParam &learner_param, int gpu_nrows)
    : param_(param), learner_param_(learner_param), gpu_batch_nrows_(gpu_nrows), row_stride_(0) {
  }

  /* Builds the sketches on the GPU for the dmatrix and returns the row stride
   * for the entire dataset */
  size_t Sketch(DMatrix *dmat, HistCutMatrix *hmat) {
    const MetaInfo &info = dmat->Info();

    row_stride_ = 0;
    sketch_container_.reset(new SketchContainer(param_, dmat));
    for (const auto &batch : dmat->GetRowBatches()) {
      this->SketchBatch(batch, info);
    }

    hmat->Init(&sketch_container_.get()->sketches_, param_.max_bin);

    return row_stride_;
  }

 private:
  std::vector<std::unique_ptr<DeviceShard>> shards_;
  const tree::TrainParam &param_;
  const LearnerTrainParam &learner_param_;
  int gpu_batch_nrows_;
  size_t row_stride_;
  std::unique_ptr<SketchContainer> sketch_container_;
};

size_t DeviceSketch
  (const tree::TrainParam &param, const LearnerTrainParam &learner_param, int gpu_batch_nrows,
   DMatrix *dmat, HistCutMatrix *hmat) {
  GPUSketcher sketcher(param, learner_param, gpu_batch_nrows);
  return sketcher.Sketch(dmat, hmat);
}

}  // namespace common
}  // namespace xgboost
