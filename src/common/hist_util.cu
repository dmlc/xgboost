#include "hip/hip_runtime.h"
/*!
 * Copyright 2018 XGBoost contributors
 */

#include <xgboost/logging.h>

#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>

#include <utility>
#include <vector>
#include <memory>
#include <mutex>

#include "hist_util.h"
#include "xgboost/host_device_vector.h"
#include "device_helpers.cuh"
#include "quantile.h"
#include "../tree/param.h"

namespace xgboost {
namespace common {

using WXQSketch = DenseCuts::WXQSketch;

__global__ void FindCutsK(WXQSketch::Entry* __restrict__ cuts,
                          const bst_float* __restrict__ data,
                          const float* __restrict__ cum_weights,
                          int nsamples,
                          int ncuts) {
  // ncuts < nsamples
  int icut = threadIdx.x + blockIdx.x * blockDim.x;
  if (icut >= ncuts) {
    return;
  }
  int isample = 0;
  if (icut == 0) {
    isample = 0;
  } else if (icut == ncuts - 1) {
    isample = nsamples - 1;
  } else {
    bst_float rank = cum_weights[nsamples - 1] / static_cast<float>(ncuts - 1)
        * static_cast<float>(icut);
    // -1 is used because cum_weights is an inclusive sum
    isample = dh::UpperBound(cum_weights, nsamples, rank);
    isample = max(0, min(isample, nsamples - 1));
  }
  // repeated values will be filtered out on the CPU
  bst_float rmin = isample > 0 ? cum_weights[isample - 1] : 0;
  bst_float rmax = cum_weights[isample];
  cuts[icut] = WXQSketch::Entry(rmin, rmax, rmax - rmin, data[isample]);
}

// predictate for thrust filtering that returns true if the element is not a NaN
struct IsNotNaN {
  __device__ bool operator()(float a) const { return !isnan(a); }
};

__global__ void UnpackFeaturesK(float* __restrict__ fvalues,
                                float* __restrict__ feature_weights,
                                const size_t* __restrict__ row_ptrs,
                                const float* __restrict__ weights,
                                Entry* entries,
                                size_t nrows_array,
                                size_t row_begin_ptr,
                                size_t nrows) {
  size_t irow = threadIdx.x + size_t(blockIdx.x) * blockDim.x;
  if (irow >= nrows) {
    return;
  }
  size_t row_length = row_ptrs[irow + 1] - row_ptrs[irow];
  int icol = threadIdx.y + blockIdx.y * blockDim.y;
  if (icol >= row_length) {
    return;
  }
  Entry entry = entries[row_ptrs[irow] - row_begin_ptr + icol];
  size_t ind = entry.index * nrows_array + irow;
  // if weights are present, ensure that a non-NaN value is written to weights
  // if and only if it is also written to features
  if (!isnan(entry.fvalue) && (weights == nullptr || !isnan(weights[irow]))) {
    fvalues[ind] = entry.fvalue;
    if (feature_weights != nullptr && weights != nullptr) {
      feature_weights[ind] = weights[irow];
    }
  }
}

/*!
 * \brief A container that holds the device sketches across all
 *  sparse page batches which are distributed to different devices.
 *  As sketches are aggregated by column, the mutex guards
 *  multiple devices pushing sketch summary for the same column
 *  across distinct rows.
 */
struct SketchContainer {
  std::vector<DenseCuts::WXQSketch> sketches_;  // NOLINT
  std::vector<std::mutex> col_locks_; // NOLINT
  static constexpr int kOmpNumColsParallelizeLimit = 1000;

  SketchContainer(int max_bin, DMatrix* dmat) : col_locks_(dmat->Info().num_col_) {
    const MetaInfo& info = dmat->Info();
    // Initialize Sketches for this dmatrix
    sketches_.resize(info.num_col_);
#pragma omp parallel for default(none) shared(info, max_bin) schedule(static) \
if (info.num_col_ > kOmpNumColsParallelizeLimit)  // NOLINT
    for (int icol = 0; icol < info.num_col_; ++icol) {  // NOLINT
      sketches_[icol].Init(info.num_row_, 1.0 / (8 * max_bin));
    }
  }

  // Prevent copying/assigning/moving this as its internals can't be assigned/copied/moved
  SketchContainer(const SketchContainer &) = delete;
  SketchContainer(const SketchContainer &&) = delete;
  SketchContainer &operator=(const SketchContainer &) = delete;
  SketchContainer &operator=(const SketchContainer &&) = delete;
};

// finds quantiles on the GPU
class GPUSketcher {
 public:
  GPUSketcher(int device, int max_bin, int gpu_nrows)
      : device_(device), max_bin_(max_bin), gpu_batch_nrows_(gpu_nrows), row_stride_(0) {}

  ~GPUSketcher() {  // NOLINT
    dh::safe_cuda(hipSetDevice(device_));
  }

  void SketchBatch(const SparsePage &batch, const MetaInfo &info) {
    n_rows_ = batch.Size();

    Init(batch, info, gpu_batch_nrows_);
    Sketch(batch, info);
    ComputeRowStride();
  }

  /* Builds the sketches on the GPU for the dmatrix and returns the row stride
   * for the entire dataset */
  size_t Sketch(DMatrix *dmat, DenseCuts *hmat) {
    const MetaInfo& info = dmat->Info();

    row_stride_ = 0;
    sketch_container_.reset(new SketchContainer(max_bin_, dmat));
    for (const auto& batch : dmat->GetBatches<SparsePage>()) {
      this->SketchBatch(batch, info);
    }

    hmat->Init(&sketch_container_->sketches_, max_bin_);
    return row_stride_;
  }

  // This needs to be public because of the __device__ lambda.
  void ComputeRowStride() {
    // Find the row stride for this batch
    auto row_iter = row_ptrs_.begin();
    // Functor for finding the maximum row size for this batch
    auto get_size = [=] __device__(size_t row) {
      return row_iter[row + 1] - row_iter[row];
    }; // NOLINT

    auto counting = thrust::make_counting_iterator(size_t(0));
    using TransformT = thrust::transform_iterator<decltype(get_size), decltype(counting), size_t>;
    TransformT row_size_iter = TransformT(counting, get_size);
    size_t batch_row_stride =
        thrust::reduce(row_size_iter, row_size_iter + n_rows_, 0, thrust::maximum<size_t>());
    row_stride_ = std::max(row_stride_, batch_row_stride);
  }

  // This needs to be public because of the __device__ lambda.
  void FindColumnCuts(size_t batch_nrows, size_t icol) {
    size_t tmp_size = tmp_storage_.size();
    // filter out NaNs in feature values
    auto fvalues_begin = fvalues_.data() + icol * gpu_batch_nrows_;
    hipcub::DeviceSelect::If(tmp_storage_.data().get(),
                          tmp_size,
                          fvalues_begin,
                          fvalues_cur_.data(),
                          num_elements_.begin(),
                          batch_nrows,
                          IsNotNaN());
    size_t nfvalues_cur = 0;
    thrust::copy_n(num_elements_.begin(), 1, &nfvalues_cur);

    // compute cumulative weights using a prefix scan
    if (has_weights_) {
      // filter out NaNs in weights;
      // since hipcub::DeviceSelect::If performs stable filtering,
      // the weights are stored in the correct positions
      auto feature_weights_begin = feature_weights_.data() + icol * gpu_batch_nrows_;
      hipcub::DeviceSelect::If(tmp_storage_.data().get(),
                            tmp_size,
                            feature_weights_begin,
                            weights_.data().get(),
                            num_elements_.begin(),
                            batch_nrows,
                            IsNotNaN());

      // sort the values and weights
      hipcub::DeviceRadixSort::SortPairs(tmp_storage_.data().get(),
                                      tmp_size,
                                      fvalues_cur_.data().get(),
                                      fvalues_begin.get(),
                                      weights_.data().get(),
                                      weights2_.data().get(),
                                      nfvalues_cur);

      // sum the weights to get cumulative weight values
      hipcub::DeviceScan::InclusiveSum(tmp_storage_.data().get(),
                                    tmp_size,
                                    weights2_.begin(),
                                    weights_.begin(),
                                    nfvalues_cur);
    } else {
      // sort the batch values
      hipcub::DeviceRadixSort::SortKeys(tmp_storage_.data().get(),
                                     tmp_size,
                                     fvalues_cur_.data().get(),
                                     fvalues_begin.get(),
                                     nfvalues_cur);

      // fill in cumulative weights with counting iterator
      thrust::copy_n(thrust::make_counting_iterator(1), nfvalues_cur, weights_.begin());
    }

    // remove repeated items and sum the weights across them;
    // non-negative weights are assumed
    hipcub::DeviceReduce::ReduceByKey(tmp_storage_.data().get(),
                                   tmp_size,
                                   fvalues_begin,
                                   fvalues_cur_.begin(),
                                   weights_.begin(),
                                   weights2_.begin(),
                                   num_elements_.begin(),
                                   thrust::maximum<bst_float>(),
                                   nfvalues_cur);
    size_t n_unique = 0;
    thrust::copy_n(num_elements_.begin(), 1, &n_unique);

    // extract cuts
    n_cuts_cur_[icol] = std::min(n_cuts_, n_unique);
    // if less elements than cuts: copy all elements with their weights
    if (n_cuts_ > n_unique) {
      float* weights2_ptr = weights2_.data().get();
      float* fvalues_ptr = fvalues_cur_.data().get();
      WXQSketch::Entry* cuts_ptr = cuts_d_.data().get() + icol * n_cuts_;
      dh::LaunchN(device_, n_unique, [=]__device__(size_t i) {
        bst_float rmax = weights2_ptr[i];
        bst_float rmin = i > 0 ? weights2_ptr[i - 1] : 0;
        cuts_ptr[i] = WXQSketch::Entry(rmin, rmax, rmax - rmin, fvalues_ptr[i]);
      });
    } else if (n_cuts_cur_[icol] > 0) {
      // if more elements than cuts: use binary search on cumulative weights
      uint32_t constexpr kBlockThreads = 256;
      uint32_t const kGrids = common::DivRoundUp(n_cuts_cur_[icol], kBlockThreads);
      dh::LaunchKernel {kGrids, kBlockThreads} (
          FindCutsK,
          cuts_d_.data().get() + icol * n_cuts_,
          fvalues_cur_.data().get(),
          weights2_.data().get(),
          n_unique,
          n_cuts_cur_[icol]);
      dh::safe_cuda(hipGetLastError());  // NOLINT
    }
  }

 private:
  void Init(const SparsePage& row_batch, const MetaInfo& info, int gpu_batch_nrows) {
    num_cols_ = info.num_col_;
    has_weights_ = info.weights_.Size() > 0;

    // find the batch size
    if (gpu_batch_nrows == 0) {
      // By default, use no more than 1/16th of GPU memory
      gpu_batch_nrows_ = dh::TotalMemory(device_) / (16 * num_cols_ * sizeof(Entry));
    } else if (gpu_batch_nrows == -1) {
      gpu_batch_nrows_ = n_rows_;
    } else {
      gpu_batch_nrows_ = gpu_batch_nrows;
    }
    if (gpu_batch_nrows_ > n_rows_) {
      gpu_batch_nrows_ = n_rows_;
    }

    constexpr int kFactor = 8;
    double eps = 1.0 / (kFactor * max_bin_);
    size_t dummy_nlevel;
    WXQSketch::LimitSizeLevel(gpu_batch_nrows_, eps, &dummy_nlevel, &n_cuts_);

    // allocate necessary GPU buffers
    dh::safe_cuda(hipSetDevice(device_));

    entries_.resize(gpu_batch_nrows_ * num_cols_);
    fvalues_.resize(gpu_batch_nrows_ * num_cols_);
    fvalues_cur_.resize(gpu_batch_nrows_);
    cuts_d_.resize(n_cuts_ * num_cols_);
    cuts_h_.resize(n_cuts_ * num_cols_);
    weights_.resize(gpu_batch_nrows_);
    weights2_.resize(gpu_batch_nrows_);
    num_elements_.resize(1);

    if (has_weights_) {
      feature_weights_.resize(gpu_batch_nrows_ * num_cols_);
    }
    n_cuts_cur_.resize(num_cols_);

    // allocate storage for CUB algorithms; the size is the maximum of the sizes
    // required for various algorithm
    size_t tmp_size = 0, cur_tmp_size = 0;
    // size for sorting
    if (has_weights_) {
      hipcub::DeviceRadixSort::SortPairs(nullptr,
                                      cur_tmp_size,
                                      fvalues_cur_.data().get(),
                                      fvalues_.data().get(),
                                      weights_.data().get(),
                                      weights2_.data().get(),
                                      gpu_batch_nrows_);
    } else {
      hipcub::DeviceRadixSort::SortKeys(nullptr,
                                     cur_tmp_size,
                                     fvalues_cur_.data().get(),
                                     fvalues_.data().get(),
                                     gpu_batch_nrows_);
    }
    tmp_size = std::max(tmp_size, cur_tmp_size);
    // size for inclusive scan
    if (has_weights_) {
      hipcub::DeviceScan::InclusiveSum(nullptr,
                                    cur_tmp_size,
                                    weights2_.begin(),
                                    weights_.begin(),
                                    gpu_batch_nrows_);
      tmp_size = std::max(tmp_size, cur_tmp_size);
    }
    // size for reduction by key
    hipcub::DeviceReduce::ReduceByKey(nullptr,
                                   cur_tmp_size,
                                   fvalues_.begin(),
                                   fvalues_cur_.begin(),
                                   weights_.begin(),
                                   weights2_.begin(),
                                   num_elements_.begin(),
                                   thrust::maximum<bst_float>(),
                                   gpu_batch_nrows_);
    tmp_size = std::max(tmp_size, cur_tmp_size);
    // size for filtering
    hipcub::DeviceSelect::If(nullptr,
                          cur_tmp_size,
                          fvalues_.begin(),
                          fvalues_cur_.begin(),
                          num_elements_.begin(),
                          gpu_batch_nrows_,
                          IsNotNaN());
    tmp_size = std::max(tmp_size, cur_tmp_size);

    tmp_storage_.resize(tmp_size);
  }

  void Sketch(const SparsePage& row_batch, const MetaInfo& info) {
    // copy rows to the device
    dh::safe_cuda(hipSetDevice(device_));
    const auto& offset_vec = row_batch.offset.HostVector();
    row_ptrs_.resize(n_rows_ + 1);
    thrust::copy(offset_vec.data(), offset_vec.data() + n_rows_ + 1, row_ptrs_.begin());
    size_t gpu_nbatches = common::DivRoundUp(n_rows_, gpu_batch_nrows_);
    for (size_t gpu_batch = 0; gpu_batch < gpu_nbatches; ++gpu_batch) {
      SketchBatch(row_batch, info, gpu_batch);
    }
  }

  void SketchBatch(const SparsePage& row_batch, const MetaInfo& info, size_t gpu_batch) {
    // compute start and end indices
    size_t batch_row_begin = gpu_batch * gpu_batch_nrows_;
    size_t batch_row_end = std::min((gpu_batch + 1) * gpu_batch_nrows_,
                                    static_cast<size_t>(n_rows_));
    size_t batch_nrows = batch_row_end - batch_row_begin;

    const auto& offset_vec = row_batch.offset.HostVector();
    const auto& data_vec = row_batch.data.HostVector();

    size_t n_entries = offset_vec[batch_row_end] - offset_vec[batch_row_begin];
    // copy the batch to the GPU
    dh::safe_cuda(hipMemcpyAsync(entries_.data().get(),
                                  data_vec.data() + offset_vec[batch_row_begin],
                                  n_entries * sizeof(Entry),
                                  hipMemcpyDefault));
    // copy the weights if necessary
    if (has_weights_) {
      const auto& weights_vec = info.weights_.HostVector();
      dh::safe_cuda(hipMemcpyAsync(weights_.data().get(),
                                    weights_vec.data() + batch_row_begin,
                                    batch_nrows * sizeof(bst_float),
                                    hipMemcpyDefault));
    }

    // unpack the features; also unpack weights if present
    thrust::fill(fvalues_.begin(), fvalues_.end(), NAN);
    if (has_weights_) {
      thrust::fill(feature_weights_.begin(), feature_weights_.end(), NAN);
    }

    dim3 block3(16, 64, 1);
    // NOTE: This will typically support ~ 4M features - 64K*64
    dim3 grid3(common::DivRoundUp(batch_nrows, block3.x),
               common::DivRoundUp(num_cols_, block3.y), 1);
    dh::LaunchKernel {grid3, block3} (
        UnpackFeaturesK,
        fvalues_.data().get(),
        has_weights_ ? feature_weights_.data().get() : nullptr,
        row_ptrs_.data().get() + batch_row_begin,
        has_weights_ ? weights_.data().get() : nullptr, entries_.data().get(),
        gpu_batch_nrows_,
        offset_vec[batch_row_begin],
        batch_nrows);

    for (int icol = 0; icol < num_cols_; ++icol) {
      FindColumnCuts(batch_nrows, icol);
    }

    // add cuts into sketches
    thrust::copy(cuts_d_.begin(), cuts_d_.end(), cuts_h_.begin());
#pragma omp parallel for default(none) schedule(static) \
if (num_cols_ > SketchContainer::kOmpNumColsParallelizeLimit) // NOLINT
    for (int icol = 0; icol < num_cols_; ++icol) {
      WXQSketch::SummaryContainer summary;
      summary.Reserve(n_cuts_);
      summary.MakeFromSorted(&cuts_h_[n_cuts_ * icol], n_cuts_cur_[icol]);

      std::lock_guard<std::mutex> lock(sketch_container_->col_locks_[icol]);
      sketch_container_->sketches_[icol].PushSummary(summary);
    }
  }

  const int device_;
  const int max_bin_;
  int gpu_batch_nrows_;
  size_t row_stride_;
  std::unique_ptr<SketchContainer> sketch_container_;

  bst_uint n_rows_{};
  int num_cols_{0};
  size_t n_cuts_{0};
  bool has_weights_{false};

  dh::device_vector<size_t> row_ptrs_{};
  dh::device_vector<Entry> entries_{};
  dh::device_vector<bst_float> fvalues_{};
  dh::device_vector<bst_float> feature_weights_{};
  dh::device_vector<bst_float> fvalues_cur_{};
  dh::device_vector<WXQSketch::Entry> cuts_d_{};
  thrust::host_vector<WXQSketch::Entry> cuts_h_{};
  dh::device_vector<bst_float> weights_{};
  dh::device_vector<bst_float> weights2_{};
  std::vector<size_t> n_cuts_cur_{};
  dh::device_vector<size_t> num_elements_{};
  dh::device_vector<char> tmp_storage_{};
};

size_t DeviceSketch(int device,
                    int max_bin,
                    int gpu_batch_nrows,
                    DMatrix* dmat,
                    HistogramCuts* hmat) {
  GPUSketcher sketcher(device, max_bin, gpu_batch_nrows);
  // We only need to return the result in HistogramCuts container, so it is safe to
  // use a pointer of local HistogramCutsDense
  DenseCuts dense_cuts(hmat);
  auto res = sketcher.Sketch(dmat, &dense_cuts);
  return res;
}

}  // namespace common
}  // namespace xgboost
