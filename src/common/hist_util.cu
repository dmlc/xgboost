#include "hip/hip_runtime.h"
/**
 * Copyright 2018~2023 by XGBoost contributors
 */
#include <thrust/binary_search.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <xgboost/logging.h>

#include <cstddef>  // for size_t
#include <memory>
#include <mutex>
#include <utility>
#include <vector>

#include "categorical.h"
#include "cuda_context.cuh"  // for HIPContext
#include "device_helpers.cuh"
#include "hist_util.cuh"
#include "hist_util.h"
#include "quantile.h"
#include "xgboost/host_device_vector.h"

namespace xgboost::common {
constexpr float SketchContainer::kFactor;

namespace detail {
size_t RequiredSampleCutsPerColumn(int max_bins, size_t num_rows) {
  double eps = 1.0 / (WQSketch::kFactor * max_bins);
  size_t dummy_nlevel;
  size_t num_cuts;
  WQuantileSketch<bst_float, bst_float>::LimitSizeLevel(
      num_rows, eps, &dummy_nlevel, &num_cuts);
  return std::min(num_cuts, num_rows);
}

size_t RequiredSampleCuts(bst_row_t num_rows, bst_feature_t num_columns,
                          size_t max_bins, size_t nnz) {
  auto per_column = RequiredSampleCutsPerColumn(max_bins, num_rows);
  auto if_dense = num_columns * per_column;
  auto result = std::min(nnz, if_dense);
  return result;
}

size_t RequiredMemory(bst_row_t num_rows, bst_feature_t num_columns, size_t nnz,
                      size_t num_bins, bool with_weights) {
  size_t peak = 0;
  // 0. Allocate cut pointer in quantile container by increasing: n_columns + 1
  size_t total = (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 1. Copy and sort: 2 * bytes_per_element * shape
  total += BytesPerElement(with_weights) * num_rows * num_columns;
  peak = std::max(peak, total);
  // 2. Deallocate bytes_per_element * shape due to reusing memory in sort.
  total -= BytesPerElement(with_weights) * num_rows * num_columns / 2;
  // 3. Allocate colomn size scan by increasing: n_columns + 1
  total += (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 4. Allocate cut pointer by increasing: n_columns + 1
  total += (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 5. Allocate cuts: assuming rows is greater than bins: n_columns * limit_size
  total += RequiredSampleCuts(num_rows, num_bins, num_bins, nnz) * sizeof(SketchEntry);
  // 6. Deallocate copied entries by reducing: bytes_per_element * shape.
  peak = std::max(peak, total);
  total -= (BytesPerElement(with_weights) * num_rows * num_columns) / 2;
  // 7. Deallocate column size scan.
  peak = std::max(peak, total);
  total -= (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 8. Deallocate cut size scan.
  total -= (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 9. Allocate final cut values, min values, cut ptrs: std::min(rows, bins + 1) *
  //    n_columns + n_columns + n_columns + 1
  total += std::min(num_rows, num_bins) * num_columns * sizeof(float);
  total += num_columns *
           sizeof(std::remove_reference_t<decltype(
                      std::declval<HistogramCuts>().MinValues())>::value_type);
  total += (num_columns + 1) *
           sizeof(std::remove_reference_t<decltype(
                      std::declval<HistogramCuts>().Ptrs())>::value_type);
  peak = std::max(peak, total);

  return peak;
}

size_t SketchBatchNumElements(size_t sketch_batch_num_elements, bst_row_t num_rows,
                              bst_feature_t columns, size_t nnz, int device, size_t num_cuts,
                              bool has_weight) {
  auto constexpr kIntMax = static_cast<std::size_t>(std::numeric_limits<std::int32_t>::max());
#if defined(XGBOOST_USE_RMM) && XGBOOST_USE_RMM == 1
  // device available memory is not accurate when rmm is used.
  return std::min(nnz, kIntMax);
#endif  // defined(XGBOOST_USE_RMM) && XGBOOST_USE_RMM == 1

  if (sketch_batch_num_elements == 0) {
    auto required_memory = RequiredMemory(num_rows, columns, nnz, num_cuts, has_weight);
    // use up to 80% of available space
    auto avail = dh::AvailableMemory(device) * 0.8;
    if (required_memory > avail) {
      sketch_batch_num_elements = avail / BytesPerElement(has_weight);
    } else {
      sketch_batch_num_elements = std::min(num_rows * static_cast<size_t>(columns), nnz);
    }
  }

  return std::min(sketch_batch_num_elements, kIntMax);
}

void SortByWeight(dh::device_vector<float>* weights, dh::device_vector<Entry>* sorted_entries) {
  // Sort both entries and wegihts.
  dh::XGBDeviceAllocator<char> alloc;
  CHECK_EQ(weights->size(), sorted_entries->size());
  thrust::sort_by_key(thrust::cuda::par(alloc), sorted_entries->begin(), sorted_entries->end(),
                      weights->begin(), detail::EntryCompareOp());

  // Scan weights
  dh::XGBCachingDeviceAllocator<char> caching;
  thrust::inclusive_scan_by_key(
      thrust::cuda::par(caching), sorted_entries->begin(), sorted_entries->end(), weights->begin(),
      weights->begin(),
      [=] __device__(const Entry& a, const Entry& b) { return a.index == b.index; });
}

void RemoveDuplicatedCategories(int32_t device, MetaInfo const& info, Span<bst_row_t> d_cuts_ptr,
                                dh::device_vector<Entry>* p_sorted_entries,
                                dh::device_vector<float>* p_sorted_weights,
                                dh::caching_device_vector<size_t>* p_column_sizes_scan) {
  info.feature_types.SetDevice(device);
  auto d_feature_types = info.feature_types.ConstDeviceSpan();
  CHECK(!d_feature_types.empty());
  auto& column_sizes_scan = *p_column_sizes_scan;
  auto& sorted_entries = *p_sorted_entries;
  // Removing duplicated entries in categorical features.

  // We don't need to accumulate weight for duplicated entries as there's no weighted
  // sketching for categorical features, the categories are the cut values.
  dh::caching_device_vector<size_t> new_column_scan(column_sizes_scan.size());
  std::size_t n_uniques{0};
  if (p_sorted_weights) {
    using Pair = thrust::tuple<Entry, float>;
    auto d_sorted_entries = dh::ToSpan(sorted_entries);
    auto d_sorted_weights = dh::ToSpan(*p_sorted_weights);
    auto val_in_it = thrust::make_zip_iterator(d_sorted_entries.data(), d_sorted_weights.data());
    auto val_out_it = thrust::make_zip_iterator(d_sorted_entries.data(), d_sorted_weights.data());
    n_uniques = dh::SegmentedUnique(
        column_sizes_scan.data().get(), column_sizes_scan.data().get() + column_sizes_scan.size(),
        val_in_it, val_in_it + sorted_entries.size(), new_column_scan.data().get(), val_out_it,
        [=] __device__(Pair const& l, Pair const& r) {
          Entry const& le = thrust::get<0>(l);
          Entry const& re = thrust::get<0>(r);
          if (le.index == re.index && IsCat(d_feature_types, le.index)) {
            return le.fvalue == re.fvalue;
          }
          return false;
        });
    p_sorted_weights->resize(n_uniques);
  } else {
    n_uniques = dh::SegmentedUnique(
        column_sizes_scan.data().get(), column_sizes_scan.data().get() + column_sizes_scan.size(),
        sorted_entries.begin(), sorted_entries.end(), new_column_scan.data().get(),
        sorted_entries.begin(), [=] __device__(Entry const& l, Entry const& r) {
          if (l.index == r.index) {
            if (IsCat(d_feature_types, l.index)) {
              return l.fvalue == r.fvalue;
            }
          }
          return false;
        });
  }
  sorted_entries.resize(n_uniques);

  // Renew the column scan and cut scan based on categorical data.
  auto d_old_column_sizes_scan = dh::ToSpan(column_sizes_scan);
  dh::caching_device_vector<SketchContainer::OffsetT> new_cuts_size(info.num_col_ + 1);
  CHECK_EQ(new_column_scan.size(), new_cuts_size.size());
  dh::LaunchN(new_column_scan.size(),
              [=, d_new_cuts_size = dh::ToSpan(new_cuts_size),
               d_old_column_sizes_scan = dh::ToSpan(column_sizes_scan),
               d_new_columns_ptr = dh::ToSpan(new_column_scan)] __device__(size_t idx) {
                d_old_column_sizes_scan[idx] = d_new_columns_ptr[idx];
                if (idx == d_new_columns_ptr.size() - 1) {
                  return;
                }
                if (IsCat(d_feature_types, idx)) {
                  // Cut size is the same as number of categories in input.
                  d_new_cuts_size[idx] = d_new_columns_ptr[idx + 1] - d_new_columns_ptr[idx];
                } else {
                  d_new_cuts_size[idx] = d_cuts_ptr[idx + 1] - d_cuts_ptr[idx];
                }
              });
  // Turn size into ptr.
  thrust::exclusive_scan(thrust::device, new_cuts_size.cbegin(), new_cuts_size.cend(),
                         d_cuts_ptr.data());
}
}  // namespace detail

void ProcessWeightedBatch(Context const* ctx, const SparsePage& page, MetaInfo const& info,
                          std::size_t begin, std::size_t end,
                          SketchContainer* sketch_container,  // <- output sketch
                          int num_cuts_per_feature, common::Span<float const> sample_weight) {
  dh::device_vector<Entry> sorted_entries;
  if (page.data.DeviceCanRead()) {
    // direct copy if data is already on device
    auto const& d_data = page.data.ConstDevicePointer();
    sorted_entries = dh::device_vector<Entry>(d_data + begin, d_data + end);
  } else {
    const auto& h_data = page.data.ConstHostVector();
    sorted_entries = dh::device_vector<Entry>(h_data.begin() + begin, h_data.begin() + end);
  }

  bst_row_t base_rowid = page.base_rowid;

  dh::device_vector<float> entry_weight;
  auto cuctx = ctx->CUDACtx();
  if (!sample_weight.empty()) {
    // Expand sample weight into entry weight.
    CHECK_EQ(sample_weight.size(), info.num_row_);
    entry_weight.resize(sorted_entries.size());
    auto d_temp_weight = dh::ToSpan(entry_weight);
    page.offset.SetDevice(ctx->Device());
    auto row_ptrs = page.offset.ConstDeviceSpan();
    thrust::for_each_n(cuctx->CTP(), thrust::make_counting_iterator(0ul), entry_weight.size(),
                       [=] __device__(std::size_t idx) {
                         std::size_t element_idx = idx + begin;
                         std::size_t ridx = dh::SegmentId(row_ptrs, element_idx);
                         d_temp_weight[idx] = sample_weight[ridx + base_rowid];
                       });
    detail::SortByWeight(&entry_weight, &sorted_entries);
  } else {
    thrust::sort(cuctx->CTP(), sorted_entries.begin(), sorted_entries.end(),
                 detail::EntryCompareOp());
  }

  HostDeviceVector<SketchContainer::OffsetT> cuts_ptr;
  dh::caching_device_vector<size_t> column_sizes_scan;
  data::IsValidFunctor dummy_is_valid(std::numeric_limits<float>::quiet_NaN());
  auto batch_it = dh::MakeTransformIterator<data::COOTuple>(
      sorted_entries.data().get(), [] __device__(Entry const& e) -> data::COOTuple {
        return {0, e.index, e.fvalue};  // row_idx is not needed for scaning column size.
      });
  detail::GetColumnSizesScan(ctx->Ordinal(), info.num_col_, num_cuts_per_feature,
                             IterSpan{batch_it, sorted_entries.size()}, dummy_is_valid, &cuts_ptr,
                             &column_sizes_scan);
  auto d_cuts_ptr = cuts_ptr.DeviceSpan();
  if (sketch_container->HasCategorical()) {
    auto p_weight = entry_weight.empty() ? nullptr : &entry_weight;
    detail::RemoveDuplicatedCategories(ctx->Ordinal(), info, d_cuts_ptr, &sorted_entries, p_weight,
                                       &column_sizes_scan);
  }

  auto const& h_cuts_ptr = cuts_ptr.ConstHostVector();
  CHECK_EQ(d_cuts_ptr.size(), column_sizes_scan.size());

  // Add cuts into sketches
  sketch_container->Push(dh::ToSpan(sorted_entries), dh::ToSpan(column_sizes_scan), d_cuts_ptr,
                         h_cuts_ptr.back(), dh::ToSpan(entry_weight));

  sorted_entries.clear();
  sorted_entries.shrink_to_fit();
  CHECK_EQ(sorted_entries.capacity(), 0);
  CHECK_NE(cuts_ptr.Size(), 0);
}

// Unify group weight, Hessian, and sample weight into sample weight.
[[nodiscard]] Span<float const> UnifyWeight(HIPContext const* cuctx, MetaInfo const& info,
                                            common::Span<float const> hessian,
                                            HostDeviceVector<float>* p_out_weight) {
  if (hessian.empty()) {
    if (info.IsRanking() && !info.weights_.Empty()) {
      common::Span<float const> group_weight = info.weights_.ConstDeviceSpan();
      dh::device_vector<bst_group_t> group_ptr(info.group_ptr_);
      auto d_group_ptr = dh::ToSpan(group_ptr);
      CHECK_GE(d_group_ptr.size(), 2) << "Must have at least 1 group for ranking.";
      auto d_weight = info.weights_.ConstDeviceSpan();
      CHECK_EQ(d_weight.size(), d_group_ptr.size() - 1)
          << "Weight size should equal to number of groups.";
      p_out_weight->Resize(info.num_row_);
      auto d_weight_out = p_out_weight->DeviceSpan();

      thrust::for_each_n(cuctx->CTP(), thrust::make_counting_iterator(0ul), d_weight_out.size(),
                         [=] XGBOOST_DEVICE(std::size_t i) {
                           auto gidx = dh::SegmentId(d_group_ptr, i);
                           d_weight_out[i] = d_weight[gidx];
                         });
      return p_out_weight->ConstDeviceSpan();
    } else {
      return info.weights_.ConstDeviceSpan();
    }
  }

  // sketch with hessian as weight
  p_out_weight->Resize(info.num_row_);
  auto d_weight_out = p_out_weight->DeviceSpan();
  if (!info.weights_.Empty()) {
    // merge sample weight with hessian
    auto d_weight = info.weights_.ConstDeviceSpan();
    if (info.IsRanking()) {
      dh::device_vector<bst_group_t> group_ptr(info.group_ptr_);
      CHECK_EQ(hessian.size(), d_weight_out.size());
      auto d_group_ptr = dh::ToSpan(group_ptr);
      CHECK_GE(d_group_ptr.size(), 2) << "Must have at least 1 group for ranking.";
      CHECK_EQ(d_weight.size(), d_group_ptr.size() - 1)
          << "Weight size should equal to number of groups.";
      thrust::for_each_n(cuctx->CTP(), thrust::make_counting_iterator(0ul), hessian.size(),
                         [=] XGBOOST_DEVICE(std::size_t i) {
                           d_weight_out[i] = d_weight[dh::SegmentId(d_group_ptr, i)] * hessian(i);
                         });
    } else {
      CHECK_EQ(hessian.size(), info.num_row_);
      CHECK_EQ(hessian.size(), d_weight.size());
      CHECK_EQ(hessian.size(), d_weight_out.size());
      thrust::for_each_n(
          cuctx->CTP(), thrust::make_counting_iterator(0ul), hessian.size(),
          [=] XGBOOST_DEVICE(std::size_t i) { d_weight_out[i] = d_weight[i] * hessian(i); });
    }
  } else {
    // copy hessian as weight
    CHECK_EQ(d_weight_out.size(), hessian.size());
    dh::safe_cuda(hipMemcpyAsync(d_weight_out.data(), hessian.data(), hessian.size_bytes(),
                                  hipMemcpyDefault));
  }
  return d_weight_out;
}

HistogramCuts DeviceSketchWithHessian(Context const* ctx, DMatrix* p_fmat, bst_bin_t max_bin,
                                      Span<float const> hessian,
                                      std::size_t sketch_batch_num_elements) {
  auto const& info = p_fmat->Info();
  bool has_weight = !info.weights_.Empty();
  info.feature_types.SetDevice(ctx->Device());

  HostDeviceVector<float> weight;
  weight.SetDevice(ctx->Device());

  // Configure batch size based on available memory
  std::size_t num_cuts_per_feature = detail::RequiredSampleCutsPerColumn(max_bin, info.num_row_);
  sketch_batch_num_elements = detail::SketchBatchNumElements(
      sketch_batch_num_elements, info.num_row_, info.num_col_, info.num_nonzero_, ctx->Ordinal(),
      num_cuts_per_feature, has_weight);

  HIPContext const* cuctx = ctx->CUDACtx();

  info.weights_.SetDevice(ctx->Device());
  auto d_weight = UnifyWeight(cuctx, info, hessian, &weight);

  HistogramCuts cuts;
  SketchContainer sketch_container(info.feature_types, max_bin, info.num_col_, info.num_row_,
                                   ctx->Ordinal());
  CHECK_EQ(has_weight || !hessian.empty(), !d_weight.empty());
  for (const auto& page : p_fmat->GetBatches<SparsePage>()) {
    std::size_t page_nnz = page.data.Size();
    for (auto begin = 0ull; begin < page_nnz; begin += sketch_batch_num_elements) {
      std::size_t end =
          std::min(page_nnz, static_cast<std::size_t>(begin + sketch_batch_num_elements));
      ProcessWeightedBatch(ctx, page, info, begin, end, &sketch_container, num_cuts_per_feature,
                           d_weight);
    }
  }

  sketch_container.MakeCuts(&cuts, p_fmat->Info().IsColumnSplit());
  return cuts;
}
}  // namespace xgboost::common
