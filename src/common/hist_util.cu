#include "hip/hip_runtime.h"
/*!
 * Copyright 2018~2020 XGBoost contributors
 */

#include <xgboost/logging.h>

#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

#include <memory>
#include <mutex>
#include <utility>
#include <vector>

#include "device_helpers.cuh"
#include "hist_util.h"
#include "hist_util.cuh"
#include "math.h"  // NOLINT
#include "quantile.h"
#include "xgboost/host_device_vector.h"


namespace xgboost {
namespace common {

constexpr float SketchContainer::kFactor;

namespace detail {

// Count the entries in each column and exclusive scan
void ExtractCutsSparse(int device, common::Span<SketchContainer::OffsetT const> cuts_ptr,
                       Span<Entry const> sorted_data,
                       Span<size_t const> column_sizes_scan,
                       Span<SketchEntry> out_cuts) {
  dh::LaunchN(device, out_cuts.size(), [=] __device__(size_t idx) {
    // Each thread is responsible for obtaining one cut from the sorted input
    size_t column_idx = dh::SegmentId(cuts_ptr, idx);
    size_t column_size =
        column_sizes_scan[column_idx + 1] - column_sizes_scan[column_idx];
    size_t num_available_cuts = cuts_ptr[column_idx + 1] - cuts_ptr[column_idx];
    size_t cut_idx = idx - cuts_ptr[column_idx];
    Span<Entry const> column_entries =
        sorted_data.subspan(column_sizes_scan[column_idx], column_size);
    size_t rank = (column_entries.size() * cut_idx) /
                  static_cast<float>(num_available_cuts);
    out_cuts[idx] = WQSketch::Entry(rank, rank + 1, 1,
                                    column_entries[rank].fvalue);
  });
}

void ExtractWeightedCutsSparse(int device,
                               common::Span<SketchContainer::OffsetT const> cuts_ptr,
                               Span<Entry> sorted_data,
                               Span<float> weights_scan,
                               Span<size_t> column_sizes_scan,
                               Span<SketchEntry> cuts) {
  dh::LaunchN(device, cuts.size(), [=] __device__(size_t idx) {
    // Each thread is responsible for obtaining one cut from the sorted input
    size_t column_idx = dh::SegmentId(cuts_ptr, idx);
    size_t column_size =
        column_sizes_scan[column_idx + 1] - column_sizes_scan[column_idx];
    size_t num_available_cuts = cuts_ptr[column_idx + 1] - cuts_ptr[column_idx];
    size_t cut_idx = idx - cuts_ptr[column_idx];

    Span<Entry> column_entries =
        sorted_data.subspan(column_sizes_scan[column_idx], column_size);

    Span<float> column_weights_scan =
        weights_scan.subspan(column_sizes_scan[column_idx], column_size);
    float total_column_weight = column_weights_scan.back();
    size_t sample_idx = 0;
    if (cut_idx == 0) {
      // First cut
      sample_idx = 0;
    } else if (cut_idx == num_available_cuts) {
      // Last cut
      sample_idx = column_entries.size() - 1;
    } else if (num_available_cuts == column_size) {
      // There are less samples available than our buffer
      // Take every available sample
      sample_idx = cut_idx;
    } else {
      bst_float rank = (total_column_weight * cut_idx) /
                       static_cast<float>(num_available_cuts);
      sample_idx = thrust::upper_bound(thrust::seq,
                                       column_weights_scan.begin(),
                                       column_weights_scan.end(),
                                       rank) -
                   column_weights_scan.begin();
      sample_idx =
          max(static_cast<size_t>(0),
              min(sample_idx, column_entries.size() - 1));
    }
    // repeated values will be filtered out later.
    bst_float rmin = sample_idx > 0 ? column_weights_scan[sample_idx - 1] : 0.0f;
    bst_float rmax = column_weights_scan[sample_idx];
    cuts[idx] = WQSketch::Entry(rmin, rmax, rmax - rmin,
                                column_entries[sample_idx].fvalue);
  });
}

size_t RequiredSampleCutsPerColumn(int max_bins, size_t num_rows) {
  double eps = 1.0 / (WQSketch::kFactor * max_bins);
  size_t dummy_nlevel;
  size_t num_cuts;
  WQuantileSketch<bst_float, bst_float>::LimitSizeLevel(
      num_rows, eps, &dummy_nlevel, &num_cuts);
  return std::min(num_cuts, num_rows);
}

size_t RequiredSampleCuts(bst_row_t num_rows, bst_feature_t num_columns,
                          size_t max_bins, size_t nnz) {
  auto per_column = RequiredSampleCutsPerColumn(max_bins, num_rows);
  auto if_dense = num_columns * per_column;
  auto result = std::min(nnz, if_dense);
  return result;
}

size_t RequiredMemory(bst_row_t num_rows, bst_feature_t num_columns, size_t nnz,
                      size_t num_bins, bool with_weights) {
  size_t peak = 0;
  // 0. Allocate cut pointer in quantile container by increasing: n_columns + 1
  size_t total = (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 1. Copy and sort: 2 * bytes_per_element * shape
  total += BytesPerElement(with_weights) * num_rows * num_columns;
  peak = std::max(peak, total);
  // 2. Deallocate bytes_per_element * shape due to reusing memory in sort.
  total -= BytesPerElement(with_weights) * num_rows * num_columns / 2;
  // 3. Allocate colomn size scan by increasing: n_columns + 1
  total += (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 4. Allocate cut pointer by increasing: n_columns + 1
  total += (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 5. Allocate cuts: assuming rows is greater than bins: n_columns * limit_size
  total += RequiredSampleCuts(num_rows, num_bins, num_bins, nnz) * sizeof(SketchEntry);
  // 6. Deallocate copied entries by reducing: bytes_per_element * shape.
  peak = std::max(peak, total);
  total -= (BytesPerElement(with_weights) * num_rows * num_columns) / 2;
  // 7. Deallocate column size scan.
  peak = std::max(peak, total);
  total -= (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 8. Deallocate cut size scan.
  total -= (num_columns + 1) * sizeof(SketchContainer::OffsetT);
  // 9. Allocate final cut values, min values, cut ptrs: std::min(rows, bins + 1) *
  //    n_columns + n_columns + n_columns + 1
  total += std::min(num_rows, num_bins) * num_columns * sizeof(float);
  total += num_columns *
           sizeof(std::remove_reference_t<decltype(
                      std::declval<HistogramCuts>().MinValues())>::value_type);
  total += (num_columns + 1) *
           sizeof(std::remove_reference_t<decltype(
                      std::declval<HistogramCuts>().Ptrs())>::value_type);
  peak = std::max(peak, total);

  return peak;
}

size_t SketchBatchNumElements(size_t sketch_batch_num_elements,
                              bst_row_t num_rows, bst_feature_t columns,
                              size_t nnz, int device,
                              size_t num_cuts, bool has_weight) {
  if (sketch_batch_num_elements == 0) {
    auto required_memory = RequiredMemory(num_rows, columns, nnz, num_cuts, has_weight);
    // use up to 80% of available space
    auto avail = dh::AvailableMemory(device) * 0.8;
    if (required_memory > avail) {
      sketch_batch_num_elements = avail / BytesPerElement(has_weight);
    } else {
      sketch_batch_num_elements = std::min(num_rows * static_cast<size_t>(columns), nnz);
    }
  }
  return sketch_batch_num_elements;
}

void SortByWeight(dh::XGBCachingDeviceAllocator<char>* alloc,
                  dh::caching_device_vector<float>* weights,
                  dh::caching_device_vector<Entry>* sorted_entries) {
  // Sort both entries and wegihts.
  thrust::sort_by_key(thrust::cuda::par(*alloc), sorted_entries->begin(),
                      sorted_entries->end(), weights->begin(),
                      detail::EntryCompareOp());

  // Scan weights
  thrust::inclusive_scan_by_key(thrust::cuda::par(*alloc),
                                sorted_entries->begin(), sorted_entries->end(),
                                weights->begin(), weights->begin(),
                                [=] __device__(const Entry& a, const Entry& b) {
                                  return a.index == b.index;
                                });
}
}  // namespace detail

void ProcessBatch(int device, const SparsePage &page, size_t begin, size_t end,
                  SketchContainer *sketch_container, int num_cuts_per_feature,
                  size_t num_columns) {
  dh::XGBCachingDeviceAllocator<char> alloc;
  const auto& host_data = page.data.ConstHostVector();
  dh::device_vector<Entry> sorted_entries(host_data.begin() + begin,
                                                  host_data.begin() + end);
  thrust::sort(thrust::cuda::par(alloc), sorted_entries.begin(),
               sorted_entries.end(), detail::EntryCompareOp());

  HostDeviceVector<SketchContainer::OffsetT> cuts_ptr;
  dh::caching_device_vector<size_t> column_sizes_scan;
  data::IsValidFunctor dummy_is_valid(std::numeric_limits<float>::quiet_NaN());
  auto batch_it = dh::MakeTransformIterator<data::COOTuple>(
      sorted_entries.data().get(),
      [] __device__(Entry const &e) -> data::COOTuple {
        return {0, e.index, e.fvalue};  // row_idx is not needed for scanning column size.
      });
  detail::GetColumnSizesScan(device, num_columns, num_cuts_per_feature,
                             batch_it, dummy_is_valid,
                             0, sorted_entries.size(),
                             &cuts_ptr, &column_sizes_scan);

  auto const& h_cuts_ptr = cuts_ptr.ConstHostVector();
  dh::caching_device_vector<SketchEntry> cuts(h_cuts_ptr.back());
  auto d_cuts_ptr = cuts_ptr.ConstDeviceSpan();

  CHECK_EQ(d_cuts_ptr.size(), column_sizes_scan.size());
  detail::ExtractCutsSparse(device, d_cuts_ptr, dh::ToSpan(sorted_entries),
                            dh::ToSpan(column_sizes_scan), dh::ToSpan(cuts));

  // add cuts into sketches
  sorted_entries.clear();
  sorted_entries.shrink_to_fit();
  CHECK_EQ(sorted_entries.capacity(), 0);
  CHECK_NE(cuts_ptr.Size(), 0);
  sketch_container->Push(cuts_ptr.ConstDeviceSpan(), &cuts);
}

void ProcessWeightedBatch(int device, const SparsePage& page,
                          Span<const float> weights, size_t begin, size_t end,
                          SketchContainer* sketch_container, int num_cuts_per_feature,
                          size_t num_columns,
                          bool is_ranking, Span<bst_group_t const> d_group_ptr) {
  dh::XGBCachingDeviceAllocator<char> alloc;
  const auto& host_data = page.data.ConstHostVector();
  dh::caching_device_vector<Entry> sorted_entries(host_data.begin() + begin,
                                                  host_data.begin() + end);

  // Binary search to assign weights to each element
  dh::caching_device_vector<float> temp_weights(sorted_entries.size());
  auto d_temp_weights = temp_weights.data().get();
  page.offset.SetDevice(device);
  auto row_ptrs = page.offset.ConstDeviceSpan();
  size_t base_rowid = page.base_rowid;
  if (is_ranking) {
    CHECK_GE(d_group_ptr.size(), 2)
        << "Must have at least 1 group for ranking.";
    CHECK_EQ(weights.size(), d_group_ptr.size() - 1)
        << "Weight size should equal to number of groups.";
    dh::LaunchN(device, temp_weights.size(), [=] __device__(size_t idx) {
        size_t element_idx = idx + begin;
        size_t ridx = dh::SegmentId(row_ptrs, element_idx);
        bst_group_t group_idx = dh::SegmentId(d_group_ptr, ridx + base_rowid);
        d_temp_weights[idx] = weights[group_idx];
      });
  } else {
    dh::LaunchN(device, temp_weights.size(), [=] __device__(size_t idx) {
        size_t element_idx = idx + begin;
        size_t ridx = dh::SegmentId(row_ptrs, element_idx);
        d_temp_weights[idx] = weights[ridx + base_rowid];
      });
  }
  detail::SortByWeight(&alloc, &temp_weights, &sorted_entries);

  HostDeviceVector<SketchContainer::OffsetT> cuts_ptr;
  dh::caching_device_vector<size_t> column_sizes_scan;
  data::IsValidFunctor dummy_is_valid(std::numeric_limits<float>::quiet_NaN());
  auto batch_it = dh::MakeTransformIterator<data::COOTuple>(
      sorted_entries.data().get(),
      [] __device__(Entry const &e) -> data::COOTuple {
        return {0, e.index, e.fvalue};  // row_idx is not needed for scaning column size.
      });
  detail::GetColumnSizesScan(device, num_columns, num_cuts_per_feature,
                             batch_it, dummy_is_valid,
                             0, sorted_entries.size(),
                             &cuts_ptr, &column_sizes_scan);

  auto const& h_cuts_ptr = cuts_ptr.ConstHostVector();
  dh::caching_device_vector<SketchEntry> cuts(h_cuts_ptr.back());
  auto d_cuts_ptr = cuts_ptr.ConstDeviceSpan();

  // Extract cuts
  detail::ExtractWeightedCutsSparse(device, d_cuts_ptr,
                                    dh::ToSpan(sorted_entries),
                                    dh::ToSpan(temp_weights),
                                    dh::ToSpan(column_sizes_scan),
                                    dh::ToSpan(cuts));

  // add cuts into sketches
  sketch_container->Push(cuts_ptr.ConstDeviceSpan(), &cuts);
}

HistogramCuts DeviceSketch(int device, DMatrix* dmat, int max_bins,
                           size_t sketch_batch_num_elements) {
  // Configure batch size based on available memory
  bool has_weights = dmat->Info().weights_.Size() > 0;
  size_t num_cuts_per_feature =
      detail::RequiredSampleCutsPerColumn(max_bins, dmat->Info().num_row_);
  sketch_batch_num_elements = detail::SketchBatchNumElements(
      sketch_batch_num_elements,
      dmat->Info().num_row_,
      dmat->Info().num_col_,
      dmat->Info().num_nonzero_,
      device, num_cuts_per_feature, has_weights);

  HistogramCuts cuts;
  SketchContainer sketch_container(max_bins, dmat->Info().num_col_,
                                   dmat->Info().num_row_, device);

  dmat->Info().weights_.SetDevice(device);
  for (const auto& batch : dmat->GetBatches<SparsePage>()) {
    size_t batch_nnz = batch.data.Size();
    auto const& info = dmat->Info();
    for (auto begin = 0ull; begin < batch_nnz; begin += sketch_batch_num_elements) {
      size_t end = std::min(batch_nnz, size_t(begin + sketch_batch_num_elements));
      if (has_weights) {
        bool is_ranking = HostSketchContainer::UseGroup(dmat->Info());
        dh::caching_device_vector<uint32_t> groups(info.group_ptr_.cbegin(),
                                                   info.group_ptr_.cend());
        ProcessWeightedBatch(
            device, batch, dmat->Info().weights_.ConstDeviceSpan(), begin, end,
            &sketch_container,
            num_cuts_per_feature,
            dmat->Info().num_col_,
            is_ranking, dh::ToSpan(groups));
      } else {
        ProcessBatch(device, batch, begin, end, &sketch_container, num_cuts_per_feature,
                     dmat->Info().num_col_);
      }
    }
  }
  sketch_container.MakeCuts(&cuts);
  return cuts;
}
}  // namespace common
}  // namespace xgboost
