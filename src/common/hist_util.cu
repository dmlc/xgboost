#include "hip/hip_runtime.h"
/*!
 * Copyright 2018 XGBoost contributors
 */

#include "./hist_util.h"

#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>

#include <utility>
#include <vector>
#include <memory>
#include <mutex>

#include "../tree/param.h"
#include "./host_device_vector.h"
#include "./device_helpers.cuh"
#include "./quantile.h"

namespace xgboost {
namespace common {

using WXQSketch = HistCutMatrix::WXQSketch;

__global__ void FindCutsK
(WXQSketch::Entry* __restrict__ cuts, const bst_float* __restrict__ data,
 const float* __restrict__ cum_weights, int nsamples, int ncuts) {
  // ncuts < nsamples
  int icut = threadIdx.x + blockIdx.x * blockDim.x;
  if (icut >= ncuts) {
    return;
  }
  WXQSketch::Entry v;
  int isample = 0;
  if (icut == 0) {
    isample = 0;
  } else if (icut == ncuts - 1) {
    isample = nsamples - 1;
  } else {
    bst_float rank = cum_weights[nsamples - 1] / static_cast<float>(ncuts - 1)
      * static_cast<float>(icut);
    // -1 is used because cum_weights is an inclusive sum
    isample = dh::UpperBound(cum_weights, nsamples, rank);
    isample = max(0, min(isample, nsamples - 1));
  }
  // repeated values will be filtered out on the CPU
  bst_float rmin = isample > 0 ? cum_weights[isample - 1] : 0;
  bst_float rmax = cum_weights[isample];
  cuts[icut] = WXQSketch::Entry(rmin, rmax, rmax - rmin, data[isample]);
}

// predictate for thrust filtering that returns true if the element is not a NaN
struct IsNotNaN {
  __device__ bool operator()(float a) const { return !isnan(a); }
};

__global__ void UnpackFeaturesK
(float* __restrict__ fvalues, float* __restrict__ feature_weights,
 const size_t* __restrict__ row_ptrs, const float* __restrict__ weights,
 Entry* entries, size_t nrows_array, int ncols, size_t row_begin_ptr,
 size_t nrows) {
  size_t irow = threadIdx.x + size_t(blockIdx.x) * blockDim.x;
  if (irow >= nrows) {
    return;
  }
  size_t row_length = row_ptrs[irow + 1] - row_ptrs[irow];
  int icol = threadIdx.y + blockIdx.y * blockDim.y;
  if (icol >= row_length) {
    return;
  }
  Entry entry = entries[row_ptrs[irow] - row_begin_ptr + icol];
  size_t ind = entry.index * nrows_array + irow;
  // if weights are present, ensure that a non-NaN value is written to weights
  // if and only if it is also written to features
  if (!isnan(entry.fvalue) && (weights == nullptr || !isnan(weights[irow]))) {
    fvalues[ind] = entry.fvalue;
    if (feature_weights != nullptr && weights != nullptr) {
      feature_weights[ind] = weights[irow];
    }
  }
}

/*!
 * \brief A container that holds the device sketches across all
 *  sparse page batches which are distributed to different devices.
 *  As sketches are aggregated by column, the mutex guards
 *  multiple devices pushing sketch summary for the same column
 *  across distinct rows.
 */
struct SketchContainer {
  std::vector<HistCutMatrix::WXQSketch> sketches_;  // NOLINT
  std::vector<std::unique_ptr<std::mutex>> col_locks_; // NOLINT
};

// finds quantiles on the GPU
struct GPUSketcher {
  // manage memory for a single GPU
  class DeviceShard {
    int device_;
    bst_uint row_begin_;  // The row offset for this shard
    bst_uint row_end_;
    bst_uint n_rows_;
    int num_cols_{0};
    size_t n_cuts_{0};
    size_t gpu_batch_nrows_{0};
    bool has_weights_{false};
    size_t row_stride_{0};

    tree::TrainParam param_;
    SketchContainer *sketch_container_;
    thrust::device_vector<size_t> row_ptrs_;
    thrust::device_vector<Entry> entries_;
    thrust::device_vector<bst_float> fvalues_;
    thrust::device_vector<bst_float> feature_weights_;
    thrust::device_vector<bst_float> fvalues_cur_;
    thrust::device_vector<WXQSketch::Entry> cuts_d_;
    thrust::host_vector<WXQSketch::Entry> cuts_h_;
    thrust::device_vector<bst_float> weights_;
    thrust::device_vector<bst_float> weights2_;
    std::vector<size_t> n_cuts_cur_;
    thrust::device_vector<size_t> num_elements_;
    thrust::device_vector<char> tmp_storage_;

   public:
    DeviceShard(int device, bst_uint row_begin, bst_uint row_end,
                tree::TrainParam param, SketchContainer *sketch_container) :
      device_(device), row_begin_(row_begin), row_end_(row_end),
      n_rows_(row_end - row_begin), param_(std::move(param)), sketch_container_(sketch_container) {
    }

    inline size_t GetRowStride() {
       return row_stride_;
    }

    void Init(const SparsePage& row_batch, const MetaInfo& info, int gpu_batch_nrows) {
      num_cols_ = info.num_col_;
      has_weights_ = info.weights_.Size() > 0;

      // find the batch size
      if (gpu_batch_nrows == 0) {
        // By default, use no more than 1/16th of GPU memory
        gpu_batch_nrows_ = dh::TotalMemory(device_) /
          (16 * num_cols_ * sizeof(Entry));
      } else if (gpu_batch_nrows == -1) {
        gpu_batch_nrows_ = n_rows_;
      } else {
        gpu_batch_nrows_ = gpu_batch_nrows;
      }
      if (gpu_batch_nrows_ > n_rows_) {
        gpu_batch_nrows_ = n_rows_;
      }

      constexpr int kFactor = 8;
      double eps = 1.0 / (kFactor * param_.max_bin);
      size_t dummy_nlevel;
      WXQSketch::LimitSizeLevel(gpu_batch_nrows_, eps, &dummy_nlevel, &n_cuts_);

      // allocate necessary GPU buffers
      dh::safe_cuda(hipSetDevice(device_));

      entries_.resize(gpu_batch_nrows_ * num_cols_);
      fvalues_.resize(gpu_batch_nrows_ * num_cols_);
      fvalues_cur_.resize(gpu_batch_nrows_);
      cuts_d_.resize(n_cuts_ * num_cols_);
      cuts_h_.resize(n_cuts_ * num_cols_);
      weights_.resize(gpu_batch_nrows_);
      weights2_.resize(gpu_batch_nrows_);
      num_elements_.resize(1);

      if (has_weights_) {
        feature_weights_.resize(gpu_batch_nrows_ * num_cols_);
      }
      n_cuts_cur_.resize(num_cols_);

      // allocate storage for CUB algorithms; the size is the maximum of the sizes
      // required for various algorithm
      size_t tmp_size = 0, cur_tmp_size = 0;
      // size for sorting
      if (has_weights_) {
        hipcub::DeviceRadixSort::SortPairs
          (nullptr, cur_tmp_size, fvalues_cur_.data().get(),
           fvalues_.data().get(), weights_.data().get(), weights2_.data().get(),
           gpu_batch_nrows_);
      } else {
        hipcub::DeviceRadixSort::SortKeys
          (nullptr, cur_tmp_size, fvalues_cur_.data().get(), fvalues_.data().get(),
           gpu_batch_nrows_);
      }
      tmp_size = std::max(tmp_size, cur_tmp_size);
      // size for inclusive scan
      if (has_weights_) {
        hipcub::DeviceScan::InclusiveSum
          (nullptr, cur_tmp_size, weights2_.begin(), weights_.begin(), gpu_batch_nrows_);
        tmp_size = std::max(tmp_size, cur_tmp_size);
      }
      // size for reduction by key
      hipcub::DeviceReduce::ReduceByKey
        (nullptr, cur_tmp_size, fvalues_.begin(),
         fvalues_cur_.begin(), weights_.begin(), weights2_.begin(),
         num_elements_.begin(), thrust::maximum<bst_float>(), gpu_batch_nrows_);
      tmp_size = std::max(tmp_size, cur_tmp_size);
      // size for filtering
      hipcub::DeviceSelect::If
        (nullptr, cur_tmp_size, fvalues_.begin(), fvalues_cur_.begin(),
         num_elements_.begin(), gpu_batch_nrows_, IsNotNaN());
      tmp_size = std::max(tmp_size, cur_tmp_size);

      tmp_storage_.resize(tmp_size);
    }

    void FindColumnCuts(size_t batch_nrows, size_t icol) {
      size_t tmp_size = tmp_storage_.size();
      // filter out NaNs in feature values
      auto fvalues_begin = fvalues_.data() + icol * gpu_batch_nrows_;
      hipcub::DeviceSelect::If
        (tmp_storage_.data().get(), tmp_size, fvalues_begin,
         fvalues_cur_.data(), num_elements_.begin(), batch_nrows, IsNotNaN());
      size_t nfvalues_cur = 0;
      thrust::copy_n(num_elements_.begin(), 1, &nfvalues_cur);

      // compute cumulative weights using a prefix scan
      if (has_weights_) {
        // filter out NaNs in weights;
        // since hipcub::DeviceSelect::If performs stable filtering,
        // the weights are stored in the correct positions
        auto feature_weights_begin = feature_weights_.data() +
          icol * gpu_batch_nrows_;
        hipcub::DeviceSelect::If
          (tmp_storage_.data().get(), tmp_size, feature_weights_begin,
           weights_.data().get(), num_elements_.begin(), batch_nrows, IsNotNaN());

        // sort the values and weights
        hipcub::DeviceRadixSort::SortPairs
          (tmp_storage_.data().get(), tmp_size, fvalues_cur_.data().get(),
           fvalues_begin.get(), weights_.data().get(), weights2_.data().get(),
           nfvalues_cur);

        // sum the weights to get cumulative weight values
        hipcub::DeviceScan::InclusiveSum
          (tmp_storage_.data().get(), tmp_size, weights2_.begin(),
           weights_.begin(), nfvalues_cur);
      } else {
        // sort the batch values
        hipcub::DeviceRadixSort::SortKeys
          (tmp_storage_.data().get(), tmp_size,
           fvalues_cur_.data().get(), fvalues_begin.get(), nfvalues_cur);

        // fill in cumulative weights with counting iterator
        thrust::copy_n(thrust::make_counting_iterator(1), nfvalues_cur,
                       weights_.begin());
      }

      // remove repeated items and sum the weights across them;
      // non-negative weights are assumed
      hipcub::DeviceReduce::ReduceByKey
        (tmp_storage_.data().get(), tmp_size, fvalues_begin,
         fvalues_cur_.begin(), weights_.begin(), weights2_.begin(),
         num_elements_.begin(), thrust::maximum<bst_float>(), nfvalues_cur);
      size_t n_unique = 0;
      thrust::copy_n(num_elements_.begin(), 1, &n_unique);

      // extract cuts
      n_cuts_cur_[icol] = std::min(n_cuts_, n_unique);
      // if less elements than cuts: copy all elements with their weights
      if (n_cuts_ > n_unique) {
        float* weights2_ptr = weights2_.data().get();
        float* fvalues_ptr = fvalues_cur_.data().get();
        WXQSketch::Entry* cuts_ptr = cuts_d_.data().get() + icol * n_cuts_;
        dh::LaunchN(device_, n_unique, [=]__device__(size_t i) {
            bst_float rmax = weights2_ptr[i];
            bst_float rmin = i > 0 ? weights2_ptr[i - 1] : 0;
            cuts_ptr[i] = WXQSketch::Entry(rmin, rmax, rmax - rmin, fvalues_ptr[i]);
          });
      } else if (n_cuts_cur_[icol] > 0) {
        // if more elements than cuts: use binary search on cumulative weights
        int block = 256;
        FindCutsK<<<dh::DivRoundUp(n_cuts_cur_[icol], block), block>>>
          (cuts_d_.data().get() + icol * n_cuts_, fvalues_cur_.data().get(),
           weights2_.data().get(), n_unique, n_cuts_cur_[icol]);
        dh::safe_cuda(hipGetLastError());  // NOLINT
      }
    }

    void SketchBatch(const SparsePage& row_batch, const MetaInfo& info,
                     size_t gpu_batch) {
      // compute start and end indices
      size_t batch_row_begin = gpu_batch * gpu_batch_nrows_;
      size_t batch_row_end = std::min((gpu_batch + 1) * gpu_batch_nrows_,
                                      static_cast<size_t>(n_rows_));
      size_t batch_nrows = batch_row_end - batch_row_begin;

      const auto& offset_vec = row_batch.offset.HostVector();
      const auto& data_vec = row_batch.data.HostVector();

      size_t n_entries = offset_vec[row_begin_ + batch_row_end] -
        offset_vec[row_begin_ + batch_row_begin];
      // copy the batch to the GPU
      dh::safe_cuda
        (hipMemcpyAsync(entries_.data().get(),
                    data_vec.data() + offset_vec[row_begin_ + batch_row_begin],
                    n_entries * sizeof(Entry), hipMemcpyDefault));
      // copy the weights if necessary
      if (has_weights_) {
        const auto& weights_vec = info.weights_.HostVector();
        dh::safe_cuda
          (hipMemcpyAsync(weights_.data().get(),
                      weights_vec.data() + row_begin_ + batch_row_begin,
                      batch_nrows * sizeof(bst_float), hipMemcpyDefault));
      }

      // unpack the features; also unpack weights if present
      thrust::fill(fvalues_.begin(), fvalues_.end(), NAN);
      if (has_weights_) {
        thrust::fill(feature_weights_.begin(), feature_weights_.end(), NAN);
      }

      dim3 block3(16, 64, 1);
      // NOTE: This will typically support ~ 4M features - 64K*64
      dim3 grid3(dh::DivRoundUp(batch_nrows, block3.x),
                 dh::DivRoundUp(num_cols_, block3.y), 1);
      UnpackFeaturesK<<<grid3, block3>>>
        (fvalues_.data().get(), has_weights_ ? feature_weights_.data().get() : nullptr,
         row_ptrs_.data().get() + batch_row_begin,
         has_weights_ ? weights_.data().get() : nullptr, entries_.data().get(),
         gpu_batch_nrows_, num_cols_,
         offset_vec[row_begin_ + batch_row_begin], batch_nrows);

      for (int icol = 0; icol < num_cols_; ++icol) {
        FindColumnCuts(batch_nrows, icol);
      }

      // add cuts into sketches
      thrust::copy(cuts_d_.begin(), cuts_d_.end(), cuts_h_.begin());
#pragma omp parallel for schedule(static)
      for (int icol = 0; icol < num_cols_; ++icol) {
        WXQSketch::SummaryContainer summary;
        summary.Reserve(n_cuts_);
        summary.MakeFromSorted(&cuts_h_[n_cuts_ * icol], n_cuts_cur_[icol]);

        std::lock_guard<std::mutex> lock(*sketch_container_->col_locks_[icol]);
        sketch_container_->sketches_[icol].PushSummary(summary);
      }
    }

    void Sketch(const SparsePage& row_batch, const MetaInfo& info) {
      // copy rows to the device
      dh::safe_cuda(hipSetDevice(device_));
      const auto& offset_vec = row_batch.offset.HostVector();
      row_ptrs_.resize(n_rows_ + 1);
      thrust::copy(offset_vec.data() + row_begin_,
                   offset_vec.data() + row_end_ + 1, row_ptrs_.begin());
      size_t gpu_nbatches = dh::DivRoundUp(n_rows_, gpu_batch_nrows_);
      for (size_t gpu_batch = 0; gpu_batch < gpu_nbatches; ++gpu_batch) {
        SketchBatch(row_batch, info, gpu_batch);
      }

      // Find the row stride for this batch
      auto row_iter = row_ptrs_.begin();
      // Functor for finding the maximum row size for this batch
      auto get_size = [=] __device__(size_t row) {
        return row_iter[row + 1] - row_iter[row];
      }; // NOLINT

      auto counting = thrust::make_counting_iterator(size_t(0));
      using TransformT = thrust::transform_iterator<decltype(get_size),
                                                    decltype(counting), size_t>;
      TransformT row_size_iter = TransformT(counting, get_size);
      row_stride_ = thrust::reduce(row_size_iter, row_size_iter + n_rows_, 0,
                                   thrust::maximum<size_t>());
    }
  };

  size_t SketchBatch(const GPUDistribution &dist, const SparsePage &batch,
                     const MetaInfo &info, SketchContainer *sketch_container) {
    // create device shards
    shards_.resize(dist.Devices().Size());
    dh::ExecuteIndexShards(&shards_, [&](int i, std::unique_ptr<DeviceShard>& shard) {
        size_t start = dist.ShardStart(batch.Size(), i);
        size_t size = dist.ShardSize(batch.Size(), i);
        shard = std::unique_ptr<DeviceShard>(
            new DeviceShard(dist.Devices().DeviceId(i), start,
                            start + size, param_, sketch_container));
      });

    // compute sketches for each shard
    dh::ExecuteIndexShards(&shards_,
                           [&](int idx, std::unique_ptr<DeviceShard>& shard) {
                             shard->Init(batch, info, gpu_batch_nrows_);
                             shard->Sketch(batch, info);
                           });
    size_t row_stride = 0;
    for (const auto &shard : shards_) {
      row_stride = std::max(row_stride, shard->GetRowStride());
    }

    return row_stride;
  }

  GPUSketcher(const tree::TrainParam &param, int gpu_nrows)
    : param_(param), gpu_batch_nrows_(gpu_nrows) {
  }

  /* Builds the sketches on the GPU */
  size_t Sketch(DMatrix *dmat, std::vector<HistCutMatrix::WXQSketch> *sketches) {
    size_t row_stride = 0;
    const MetaInfo &info = dmat->Info();

    // Initialize Sketches for this dmatrix
    SketchContainer sketch_container;
    sketch_container.sketches_.resize(info.num_col_);
    sketch_container.col_locks_.resize(info.num_col_);
#pragma omp parallel for schedule(static) if (info.num_col_ > 1000)
    for (int icol = 0; icol < info.num_col_; ++icol) {
      sketch_container.sketches_[icol].Init(info.num_row_, 1.0 / (8 * param_.max_bin));
      sketch_container.col_locks_[icol].reset(new std::mutex);
    }

    for (const auto &batch : dmat->GetRowBatches()) {
      GPUDistribution dist =
        GPUDistribution::Block(GPUSet::All(param_.gpu_id, param_.n_gpus, batch.Size()));
      size_t batch_row_stride = this->SketchBatch(dist, batch, info, &sketch_container);
      row_stride = std::max(row_stride, batch_row_stride);
    }

    sketches->swap(sketch_container.sketches_);

    return row_stride;
  }

 private:
  std::vector<std::unique_ptr<DeviceShard>> shards_;
  const tree::TrainParam &param_;
  int gpu_batch_nrows_;
};

size_t DeviceSketch
  (const tree::TrainParam &param, int gpu_batch_nrows, DMatrix *dmat,
   std::vector<HistCutMatrix::WXQSketch> *sketches) {
  GPUSketcher sketcher(param, gpu_batch_nrows);
  return sketcher.Sketch(dmat, sketches);
}

}  // namespace common
}  // namespace xgboost
