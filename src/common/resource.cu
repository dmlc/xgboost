/**
 * Copyright 2024-2025, XGBoost Contributors
 */
#include "device_helpers.cuh"  // for CurrentDevice
#include "resource.cuh"
#include "xgboost/string_view.h"  // for StringView

namespace xgboost::common {
CudaMmapResource::CudaMmapResource(StringView path, std::size_t offset, std::size_t length)
    : ResourceHandler{kCudaMmap},
      handle_{detail::OpenMmap(std::string{path}, offset, length),
              [](MMAPFile* handle) {
                // Don't close the mmap while CUDA kernel is running.
                if (handle) {
                  dh::DefaultStream().Sync();
                }
                detail::CloseMmap(handle);
              }},
      n_{length} {
  auto device = dh::CurrentDevice();
#if (CUDA_VERSION / 1000) >= 13
  hipMemLocation loc;
  loc.type = hipMemLocationTypeDevice;
  loc.id = device;
#else
  auto loc = device;
#endif  // (CUDA_VERSION / 1000) >= 13
  dh::safe_cuda(
      hipMemAdvise(handle_->base_ptr, handle_->base_size, hipMemAdviseSetReadMostly, loc));
  dh::safe_cuda(
      hipMemAdvise(handle_->base_ptr, handle_->base_size, hipMemAdviseSetPreferredLocation, loc));
  dh::safe_cuda(
      hipMemAdvise(handle_->base_ptr, handle_->base_size, hipMemAdviseSetAccessedBy, loc));
#if (CUDA_VERSION / 1000) >= 13
  dh::safe_cuda(
      hipMemPrefetchAsync(handle_->base_ptr, handle_->base_size, loc, 0, dh::DefaultStream()));
#else
  dh::safe_cuda(
      hipMemPrefetchAsync(handle_->base_ptr, handle_->base_size, device, dh::DefaultStream()));
#endif  // (CUDA_VERSION / 1000) >= 13
}

[[nodiscard]] void* CudaMmapResource::Data() {
  if (!handle_) {
    return nullptr;
  }
  return this->handle_->Data();
}

[[nodiscard]] std::size_t CudaMmapResource::Size() const { return n_; }

CudaMmapResource::~CudaMmapResource() noexcept(false) = default;

PrivateCudaMmapConstStream::~PrivateCudaMmapConstStream() noexcept(false) = default;
}  // namespace xgboost::common
