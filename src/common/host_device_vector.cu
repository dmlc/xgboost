/*!
 * Copyright 2017 XGBoost contributors
 */

#include "./host_device_vector.h"
#include <thrust/fill.h>
#include <xgboost/data.h>
#include "./device_helpers.cuh"


namespace xgboost {


template <typename T>
struct HostDeviceVectorImpl {
  struct DeviceShard {
    DeviceShard()
      : index_(-1), proper_size_(0), device_(-1), start_(0), on_d_(false), cached_size_(~0),
        vec_(nullptr) {}

    void Init(HostDeviceVectorImpl<T>* vec, int device) {
      if (vec_ == nullptr) { vec_ = vec; }
      CHECK_EQ(vec, vec_);
      device_ = device;
      index_ = vec_->distribution_.devices_.Index(device);
      LazyResize(vec_->Size());
      on_d_ = !vec_->on_h_;
    }

    void ScatterFrom(const T* begin) {
      // TODO(canonizer): avoid full copy of host data
      LazySyncDevice();
      dh::safe_cuda(hipSetDevice(device_));
      dh::safe_cuda(hipMemcpy(data_.data().get(), begin + start_,
                               data_.size() * sizeof(T), hipMemcpyDefault));
    }

    void GatherTo(thrust::device_ptr<T> begin) {
      LazySyncDevice();
      dh::safe_cuda(hipSetDevice(device_));
      dh::safe_cuda(hipMemcpy(begin.get() + start_, data_.data().get(),
                               proper_size_ * sizeof(T), hipMemcpyDefault));
    }

    void Fill(T v) {
      // TODO(canonizer): avoid full copy of host data
      LazySyncDevice();
      dh::safe_cuda(hipSetDevice(device_));
      thrust::fill(data_.begin(), data_.end(), v);
    }

    void Copy(DeviceShard* other) {
      // TODO(canonizer): avoid full copy of host data for this (but not for other)
      LazySyncDevice();
      other->LazySyncDevice();
      dh::safe_cuda(hipSetDevice(device_));
      dh::safe_cuda(hipMemcpy(data_.data().get(), other->data_.data().get(),
                               data_.size() * sizeof(T), hipMemcpyDefault));
    }

    void LazySyncHost() {
      dh::safe_cuda(hipSetDevice(device_));
      dh::safe_cuda(hipMemcpy(vec_->data_h_.data() + start_,
                               data_.data().get(),  proper_size_ * sizeof(T),
                               hipMemcpyDeviceToHost));
      on_d_ = false;
    }

    void LazyResize(size_t new_size) {
      if (new_size == cached_size_) { return; }
      // resize is required
      int ndevices = vec_->distribution_.devices_.Size();
      start_ = vec_->distribution_.ShardStart(new_size, index_);
      proper_size_ = vec_->distribution_.ShardProperSize(new_size, index_);
      size_t size_d = vec_->distribution_.ShardSize(new_size, index_);
      dh::safe_cuda(hipSetDevice(device_));
      data_.resize(size_d);
      cached_size_ = new_size;
    }

    void LazySyncDevice() {
      if (on_d_) { return; }
      // data is on the host
      size_t size_h = vec_->data_h_.size();
      LazyResize(size_h);
      dh::safe_cuda(hipSetDevice(device_));
      dh::safe_cuda(
          hipMemcpy(data_.data().get(), vec_->data_h_.data() + start_,
                     data_.size() * sizeof(T), hipMemcpyHostToDevice));
      on_d_ = true;
      // this may cause a race condition if LazySyncDevice() is called
      // from multiple threads in parallel;
      // however, the race condition is benign, and will not cause problems
      vec_->on_h_ = false;
      vec_->size_d_ = size_h;
    }

    int index_;
    int device_;
    thrust::device_vector<T> data_;
    // cached vector size
    size_t cached_size_;
    size_t start_;
    // size of the portion to copy back to the host
    size_t proper_size_;
    // true if there is an up-to-date copy of data on device, false otherwise
    bool on_d_;
    HostDeviceVectorImpl<T>* vec_;
  };

  HostDeviceVectorImpl(size_t size, T v, GPUDistribution distribution)
    : distribution_(distribution), on_h_(distribution.IsEmpty()), size_d_(0) {
    if (!distribution_.IsEmpty()) {
      size_d_ = size;
      InitShards();
      Fill(v);
    } else {
      data_h_.resize(size, v);
    }
  }

  // Init can be std::vector<T> or std::initializer_list<T>
  template <class Init>
  HostDeviceVectorImpl(const Init& init, GPUDistribution distribution)
    : distribution_(distribution), on_h_(distribution.IsEmpty()), size_d_(0) {
    if (!distribution_.IsEmpty()) {
      size_d_ = init.size();
      InitShards();
      Copy(init);
    } else {
      data_h_ = init;
    }
  }

  void InitShards() {
    int ndevices = distribution_.devices_.Size();
    shards_.resize(ndevices);
    dh::ExecuteIndexShards(&shards_, [&](int i, DeviceShard& shard) {
        shard.Init(this, distribution_.devices_[i]);
      });
  }

  size_t Size() const { return on_h_ ? data_h_.size() : size_d_; }

  GPUSet Devices() const { return distribution_.devices_; }

  const GPUDistribution& Distribution() const { return distribution_; }

  T* DevicePointer(int device) {
    CHECK(distribution_.devices_.Contains(device));
    LazySyncDevice(device);
    return shards_[distribution_.devices_.Index(device)].data_.data().get();
  }

  common::Span<T> DeviceSpan(int device) {
    CHECK(devices_.Contains(device));
    LazySyncDevice(device);
    return { shards_[devices_.Index(device)].data_.data().get(),
             static_cast<typename common::Span<T>::index_type>(Size()) };
  }

  size_t DeviceSize(int device) {
    CHECK(distribution_.devices_.Contains(device));
    LazySyncDevice(device);
    return shards_[distribution_.devices_.Index(device)].data_.size();
  }

  size_t DeviceStart(int device) {
    CHECK(distribution_.devices_.Contains(device));
    LazySyncDevice(device);
    return shards_[distribution_.devices_.Index(device)].start_;
  }

  thrust::device_ptr<T> tbegin(int device) {  // NOLINT
    return thrust::device_ptr<T>(DevicePointer(device));
  }

  thrust::device_ptr<T> tend(int device) {  // NOLINT
    return tbegin(device) + DeviceSize(device);
  }

  void ScatterFrom(thrust::device_ptr<const T> begin, thrust::device_ptr<const T> end) {
    CHECK_EQ(end - begin, Size());
    if (on_h_) {
      dh::safe_cuda(hipMemcpy(data_h_.data(), begin.get(),
                               (end - begin) * sizeof(T),
                               hipMemcpyDeviceToHost));
    } else {
      dh::ExecuteShards(&shards_, [&](DeviceShard& shard) {
        shard.ScatterFrom(begin.get());
      });
    }
  }

  void GatherTo(thrust::device_ptr<T> begin, thrust::device_ptr<T> end) {
    CHECK_EQ(end - begin, Size());
    if (on_h_) {
      dh::safe_cuda(hipMemcpy(begin.get(), data_h_.data(),
                               data_h_.size() * sizeof(T),
                               hipMemcpyHostToDevice));
    } else {
      dh::ExecuteShards(&shards_, [&](DeviceShard& shard) { shard.GatherTo(begin); });
    }
  }

  void Fill(T v) {
    if (on_h_) {
      std::fill(data_h_.begin(), data_h_.end(), v);
    } else {
      dh::ExecuteShards(&shards_, [&](DeviceShard& shard) { shard.Fill(v); });
    }
  }

  void Copy(HostDeviceVectorImpl<T>* other) {
    CHECK_EQ(Size(), other->Size());
    if (on_h_ && other->on_h_) {
      std::copy(other->data_h_.begin(), other->data_h_.end(), data_h_.begin());
    } else {
      CHECK(distribution_ == other->distribution_);
      dh::ExecuteIndexShards(&shards_, [&](int i, DeviceShard& shard) {
          shard.Copy(&other->shards_[i]);
        });
    }
  }

  void Copy(const std::vector<T>& other) {
    CHECK_EQ(Size(), other.size());
    if (on_h_) {
      std::copy(other.begin(), other.end(), data_h_.begin());
    } else {
      dh::ExecuteShards(&shards_, [&](DeviceShard& shard) {
          shard.ScatterFrom(other.data());
        });
    }
  }

  void Copy(std::initializer_list<T> other) {
    CHECK_EQ(Size(), other.size());
    if (on_h_) {
      std::copy(other.begin(), other.end(), data_h_.begin());
    } else {
      dh::ExecuteShards(&shards_, [&](DeviceShard& shard) {
          shard.ScatterFrom(other.begin());
        });
    }
  }

  std::vector<T>& HostVector() {
    LazySyncHost();
    return data_h_;
  }

  void Reshard(const GPUDistribution& distribution) {
    if (distribution_ == distribution)
      return;
    CHECK(distribution_.IsEmpty());
    distribution_ = distribution;
    InitShards();
  }

  void Reshard(GPUSet new_devices) {
    if (distribution_.Devices() == new_devices)
      return;
    Reshard(GPUDistribution::Block(new_devices));
  }

  void Resize(size_t new_size, T v) {
    if (new_size == Size())
      return;
    if (distribution_.IsFixedSize()) {
      CHECK_EQ(new_size, distribution_.offsets_.back());
    }
    if (Size() == 0 && !distribution_.IsEmpty()) {
      // fast on-device resize
      on_h_ = false;
      size_d_ = new_size;
      InitShards();
      Fill(v);
    } else {
      // resize on host
      LazySyncHost();
      data_h_.resize(new_size, v);
    }
  }

  void LazySyncHost() {
    if (on_h_)
      return;
    if (data_h_.size() != size_d_)
      data_h_.resize(size_d_);
    dh::ExecuteShards(&shards_, [&](DeviceShard& shard) { shard.LazySyncHost(); });
    on_h_ = true;
  }

  void LazySyncDevice(int device) {
    GPUSet devices = distribution_.Devices();
    CHECK(devices.Contains(device));
    shards_[devices.Index(device)].LazySyncDevice();
  }

  std::vector<T> data_h_;
  bool on_h_;
  // the total size of the data stored on the devices
  size_t size_d_;
  GPUDistribution distribution_;
  std::vector<DeviceShard> shards_;
};

template <typename T>
HostDeviceVector<T>::HostDeviceVector
(size_t size, T v, GPUDistribution distribution) : impl_(nullptr) {
  impl_ = new HostDeviceVectorImpl<T>(size, v, distribution);
}

template <typename T>
HostDeviceVector<T>::HostDeviceVector
(std::initializer_list<T> init, GPUDistribution distribution) : impl_(nullptr) {
  impl_ = new HostDeviceVectorImpl<T>(init, distribution);
}

template <typename T>
HostDeviceVector<T>::HostDeviceVector
(const std::vector<T>& init, GPUDistribution distribution) : impl_(nullptr) {
  impl_ = new HostDeviceVectorImpl<T>(init, distribution);
}

template <typename T>
HostDeviceVector<T>::HostDeviceVector(const HostDeviceVector<T>& other)
  : impl_(nullptr) {
  impl_ = new HostDeviceVectorImpl<T>(*other.impl_);
}

template <typename T>
HostDeviceVector<T>& HostDeviceVector<T>::operator=
(const HostDeviceVector<T>& other) {
  if (this == &other) {
    return *this;
  }
  delete impl_;
  impl_ = new HostDeviceVectorImpl<T>(*other.impl_);
  return *this;
}

template <typename T>
HostDeviceVector<T>::~HostDeviceVector() {
  HostDeviceVectorImpl<T>* tmp = impl_;
  impl_ = nullptr;
  delete tmp;
}

template <typename T>
size_t HostDeviceVector<T>::Size() const { return impl_->Size(); }

template <typename T>
GPUSet HostDeviceVector<T>::Devices() const { return impl_->Devices(); }

template <typename T>
const GPUDistribution& HostDeviceVector<T>::Distribution() const {
  return impl_->Distribution();
}

template <typename T>
T* HostDeviceVector<T>::DevicePointer(int device) { return impl_->DevicePointer(device); }

template <typename T>
common::Span<T> HostDeviceVector<T>::DeviceSpan(int device) {
  return impl_->DeviceSpan(device);
}

const T* HostDeviceVector<T>::ConstDevicePointer(int device) const {
  return impl_->DevicePointer(device);
}

template <typename T>
size_t HostDeviceVector<T>::DeviceStart(int device) { return impl_->DeviceStart(device); }

const T* HostDeviceVector<T>::DevicePointer(int device) const {
  return impl_->DevicePointer(device);
}

template <typename T>
size_t HostDeviceVector<T>::DeviceSize(int device) const {
  return impl_->DeviceSize(device);
}

template <typename T>
thrust::device_ptr<T> HostDeviceVector<T>::tbegin(int device) {  // NOLINT
  return impl_->tbegin(device);
}

template <typename T>
thrust::device_ptr<const T> HostDeviceVector<T>::tcbegin(int device) const {  // NOLINT
  return thrust::device_ptr<const T>(impl_->tbegin(device));
}

template <typename T>
thrust::device_ptr<T> HostDeviceVector<T>::tend(int device) {  // NOLINT
  return impl_->tend(device);
}

template <typename T>
thrust::device_ptr<const T> HostDeviceVector<T>::tcend(int device) const {  // NOLINT
  return thrust::device_ptr<const T>(impl_->tend(device));
}

template <typename T>
void HostDeviceVector<T>::ScatterFrom
(thrust::device_ptr<const T> begin, thrust::device_ptr<const T> end) {
  impl_->ScatterFrom(begin, end);
}

template <typename T>
void HostDeviceVector<T>::GatherTo
(thrust::device_ptr<T> begin, thrust::device_ptr<T> end) const {
  impl_->GatherTo(begin, end);
}

template <typename T>
void HostDeviceVector<T>::Fill(T v) {
  impl_->Fill(v);
}

template <typename T>
void HostDeviceVector<T>::Copy(const HostDeviceVector<T>& other) {
  impl_->Copy(other.impl_);
}

template <typename T>
void HostDeviceVector<T>::Copy(const std::vector<T>& other) {
  impl_->Copy(other);
}

template <typename T>
void HostDeviceVector<T>::Copy(std::initializer_list<T> other) {
  impl_->Copy(other);
}

template <typename T>
std::vector<T>& HostDeviceVector<T>::HostVector() { return impl_->HostVector(); }

template <typename T>
const std::vector<T>& HostDeviceVector<T>::ConstHostVector() const {
  return impl_->HostVector();
}

template <typename T>
void HostDeviceVector<T>::Reshard(GPUSet new_devices) const {
  impl_->Reshard(new_devices);
}

template <typename T>
void HostDeviceVector<T>::Reshard(const GPUDistribution& distribution) const {
  impl_->Reshard(distribution);
}

template <typename T>
void HostDeviceVector<T>::Resize(size_t new_size, T v) {
  impl_->Resize(new_size, v);
}

// explicit instantiations are required, as HostDeviceVector isn't header-only
template class HostDeviceVector<bst_float>;
template class HostDeviceVector<GradientPair>;
template class HostDeviceVector<unsigned int>;
template class HostDeviceVector<int>;
template class HostDeviceVector<Entry>;
template class HostDeviceVector<size_t>;

}  // namespace xgboost
