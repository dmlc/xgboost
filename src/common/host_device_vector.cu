/*!
 * Copyright 2017 XGBoost contributors
 */
#include "./host_device_vector.h"
#include "./device_helpers.cuh"

namespace xgboost {

template <typename T>
struct HostDeviceVectorImpl {
  HostDeviceVectorImpl(size_t size, int device)
    : device_(device), on_d_(device >= 0) {
    if (on_d_) {
      dh::safe_cuda(hipSetDevice(device_));
      data_d_.resize(size);
    } else {
      data_h_.resize(size);
    }
  }
  HostDeviceVectorImpl(const HostDeviceVectorImpl<T>&) = delete;
  HostDeviceVectorImpl(HostDeviceVectorImpl<T>&&) = delete;
  void operator=(const HostDeviceVectorImpl<T>&) = delete;
  void operator=(HostDeviceVectorImpl<T>&&) = delete;

  size_t size() const { return on_d_ ? data_d_.size() : data_h_.size(); }

  int device() const { return device_; }

  T* ptr_d(int device) {
    lazy_sync_device(device);
    return data_d_.data().get();
  }
  thrust::device_ptr<T> tbegin(int device) {
    return thrust::device_ptr<T>(ptr_d(device));
  }
  thrust::device_ptr<T> tend(int device) {
    auto begin = tbegin(device);
    return begin + size();
  }
  std::vector<T>& data_h() {
    lazy_sync_host();
    return data_h_;
  }
  void resize(size_t new_size, int new_device) {
    if (new_size == this->size() && new_device == device_)
      return;
    device_ = new_device;
    // if !on_d_, but the data size is 0 and the device is set,
    // resize the data on device instead
    if (!on_d_ && (data_h_.size() > 0 || device_ == -1)) {
      data_h_.resize(new_size);
    } else {
      dh::safe_cuda(hipSetDevice(device_));
      data_d_.resize(new_size);
      on_d_ = true;
    }
  }

  void lazy_sync_host() {
    if (!on_d_)
      return;
    if (data_h_.size() != this->size())
      data_h_.resize(this->size());
    dh::safe_cuda(hipSetDevice(device_));
    thrust::copy(data_d_.begin(), data_d_.end(), data_h_.begin());
    on_d_ = false;
  }

  void lazy_sync_device(int device) {
    if (on_d_)
      return;
    if (device != device_) {
      CHECK_EQ(device_, -1);
      device_ = device;
    }
    if (data_d_.size() != this->size()) {
      dh::safe_cuda(hipSetDevice(device_));
      data_d_.resize(this->size());
    }
    dh::safe_cuda(hipSetDevice(device_));
    thrust::copy(data_h_.begin(), data_h_.end(), data_d_.begin());
    on_d_ = true;
  }

  std::vector<T> data_h_;
  thrust::device_vector<T> data_d_;
  // true if there is an up-to-date copy of data on device, false otherwise
  bool on_d_;
  int device_;
};

template <typename T>
HostDeviceVector<T>::HostDeviceVector(size_t size, int device) : impl_(nullptr) {
  impl_ = new HostDeviceVectorImpl<T>(size, device);
}

template <typename T>
HostDeviceVector<T>::~HostDeviceVector() {
  HostDeviceVectorImpl<T>* tmp = impl_;
  impl_ = nullptr;
  delete tmp;
}

template <typename T>
size_t HostDeviceVector<T>::size() const { return impl_->size(); }

template <typename T>
int HostDeviceVector<T>::device() const { return impl_->device(); }

template <typename T>
T* HostDeviceVector<T>::ptr_d(int device) { return impl_->ptr_d(device); }

template <typename T>
thrust::device_ptr<T> HostDeviceVector<T>::tbegin(int device) {
  return impl_->tbegin(device);
}

template <typename T>
thrust::device_ptr<T> HostDeviceVector<T>::tend(int device) {
  return impl_->tend(device);
}

template <typename T>
std::vector<T>& HostDeviceVector<T>::data_h() { return impl_->data_h(); }

template <typename T>
void HostDeviceVector<T>::resize(size_t new_size, int new_device) {
  impl_->resize(new_size, new_device);
}

// explicit instantiations are required, as HostDeviceVector isn't header-only
template class HostDeviceVector<bst_float>;
template class HostDeviceVector<bst_gpair>;

}  // namespace xgboost
