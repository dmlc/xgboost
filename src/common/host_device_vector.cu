#include "hip/hip_runtime.h"
/*!
 * Copyright 2017 XGBoost contributors
 */

#include "./host_device_vector.h"
#include <thrust/fill.h>
#include <xgboost/data.h>
#include <algorithm>
#include <cstdint>
#include <mutex>
#include "./device_helpers.cuh"


namespace xgboost {

// the handler to call instead of hipSetDevice; only used for testing
static void (*cudaSetDeviceHandler)(int) = nullptr;  // NOLINT

void SetCudaSetDeviceHandler(void (*handler)(int)) {
  cudaSetDeviceHandler = handler;
}

// wrapper over access with useful methods
class Permissions {
  GPUAccess access_;
  explicit Permissions(GPUAccess access) : access_{access} {}

 public:
  Permissions() : access_{GPUAccess::kNone} {}
  explicit Permissions(bool perm)
    : access_(perm ? GPUAccess::kWrite : GPUAccess::kNone) {}

  bool CanRead() const { return access_ >= kRead; }
  bool CanWrite() const { return access_ == kWrite; }
  bool CanAccess(GPUAccess access) const { return access_ >= access; }
  void Grant(GPUAccess access) { access_ = std::max(access_, access); }
  void DenyComplementary(GPUAccess compl_access) {
    access_ = std::min(access_, GPUAccess::kWrite - compl_access);
  }
  Permissions Complementary() const {
    return Permissions(GPUAccess::kWrite - access_);
  }
};

template <typename T>
struct HostDeviceVectorImpl {
  struct DeviceShard {
    DeviceShard()
      : proper_size_{0}, device_{-1}, start_{0}, perm_d_{false},
        cached_size_{static_cast<size_t>(~0)}, vec_{nullptr} {}

    ~DeviceShard() {
      SetDevice();
    }

    void Init(HostDeviceVectorImpl<T>* vec, int device) {
      if (vec_ == nullptr) { vec_ = vec; }
      CHECK_EQ(vec, vec_);
      device_ = device;
      LazyResize(vec_->Size());
      perm_d_ = vec_->perm_h_.Complementary();
    }

    void Init(HostDeviceVectorImpl<T>* vec, const DeviceShard& other) {
      if (vec_ == nullptr) { vec_ = vec; }
      CHECK_EQ(vec, vec_);
      device_ = other.device_;
      cached_size_ = other.cached_size_;
      start_ = other.start_;
      proper_size_ = other.proper_size_;
      SetDevice();
      data_.resize(other.data_.size());
      perm_d_ = other.perm_d_;
    }

    void ScatterFrom(const T* begin) {
      // TODO(canonizer): avoid full copy of host data
      LazySyncDevice(GPUAccess::kWrite);
      SetDevice();
      dh::safe_cuda(hipMemcpyAsync(data_.data().get(), begin + start_,
                               data_.size() * sizeof(T), hipMemcpyDefault));
    }

    void GatherTo(thrust::device_ptr<T> begin) {
      LazySyncDevice(GPUAccess::kRead);
      SetDevice();
      dh::safe_cuda(hipMemcpyAsync(begin.get() + start_, data_.data().get(),
                               proper_size_ * sizeof(T), hipMemcpyDefault));
    }

    void Fill(T v) {
      // TODO(canonizer): avoid full copy of host data
      LazySyncDevice(GPUAccess::kWrite);
      SetDevice();
      thrust::fill(data_.begin(), data_.end(), v);
    }

    void Copy(DeviceShard* other) {
      // TODO(canonizer): avoid full copy of host data for this (but not for other)
      LazySyncDevice(GPUAccess::kWrite);
      other->LazySyncDevice(GPUAccess::kRead);
      SetDevice();
      dh::safe_cuda(hipMemcpyAsync(data_.data().get(), other->data_.data().get(),
                               data_.size() * sizeof(T), hipMemcpyDefault));
    }

    void LazySyncHost(GPUAccess access) {
      SetDevice();
      dh::safe_cuda(hipMemcpy(vec_->data_h_.data() + start_,
                               data_.data().get(),  proper_size_ * sizeof(T),
                               hipMemcpyDeviceToHost));
      perm_d_.DenyComplementary(access);
    }

    void LazyResize(size_t new_size) {
      if (new_size == cached_size_) { return; }
      // resize is required
      int ndevices = vec_->distribution_.devices_.Size();
      int device_index = vec_->distribution_.devices_.Index(device_);
      start_ = vec_->distribution_.ShardStart(new_size, device_index);
      proper_size_ = vec_->distribution_.ShardProperSize(new_size, device_index);
      // The size on this device.
      size_t size_d = vec_->distribution_.ShardSize(new_size, device_index);
      SetDevice();
      data_.resize(size_d);
      cached_size_ = new_size;
    }

    void LazySyncDevice(GPUAccess access) {
      if (perm_d_.CanAccess(access)) { return; }
      if (perm_d_.CanRead()) {
        // deny read to the host
        perm_d_.Grant(access);
        std::lock_guard<std::mutex> lock(vec_->mutex_);
        vec_->perm_h_.DenyComplementary(access);
        return;
      }
      // data is on the host
      size_t size_h = vec_->data_h_.size();
      LazyResize(size_h);
      SetDevice();
      dh::safe_cuda(
          hipMemcpy(data_.data().get(), vec_->data_h_.data() + start_,
                     data_.size() * sizeof(T), hipMemcpyHostToDevice));
      perm_d_.Grant(access);

      std::lock_guard<std::mutex> lock(vec_->mutex_);
      vec_->perm_h_.DenyComplementary(access);
      vec_->size_d_ = size_h;
    }

    void SetDevice() {
      if (cudaSetDeviceHandler == nullptr) {
        dh::safe_cuda(hipSetDevice(device_));
      } else {
        (*cudaSetDeviceHandler)(device_);
      }
    }

    T*     Raw()                    { return data_.data().get(); }
    size_t Start()            const { return start_; }
    size_t DataSize()         const { return data_.size(); }
    Permissions& Perm()             { return perm_d_; }
    Permissions const& Perm() const { return perm_d_; }

   private:
    int device_;
    dh::device_vector<T> data_;
    // cached vector size
    size_t cached_size_;
    size_t start_;
    // size of the portion to copy back to the host
    size_t proper_size_;
    Permissions perm_d_;
    HostDeviceVectorImpl<T>* vec_;
  };

  HostDeviceVectorImpl(size_t size, T v, const GPUDistribution &distribution)
    : distribution_(distribution), perm_h_(distribution.IsEmpty()), size_d_(0) {
    if (!distribution_.IsEmpty()) {
      size_d_ = size;
      InitShards();
      Fill(v);
    } else {
      data_h_.resize(size, v);
    }
  }

  // required, as a new std::mutex has to be created
  HostDeviceVectorImpl(const HostDeviceVectorImpl<T>& other)
    : data_h_(other.data_h_), perm_h_(other.perm_h_), size_d_(other.size_d_),
      distribution_(other.distribution_), mutex_() {
    shards_.resize(other.shards_.size());
    dh::ExecuteIndexShards(&shards_, [&](int i, DeviceShard& shard) {
        shard.Init(this, other.shards_.at(i));
      });
  }

  // Initializer can be std::vector<T> or std::initializer_list<T>
  template <class Initializer>
  HostDeviceVectorImpl(const Initializer& init, const GPUDistribution &distribution)
    : distribution_(distribution), perm_h_(distribution.IsEmpty()), size_d_(0) {
    if (!distribution_.IsEmpty()) {
      size_d_ = init.size();
      InitShards();
      Copy(init);
    } else {
      data_h_ = init;
    }
  }

  void InitShards() {
    int ndevices = distribution_.devices_.Size();
    shards_.resize(ndevices);
    dh::ExecuteIndexShards(&shards_, [&](int i, DeviceShard& shard) {
        shard.Init(this, distribution_.devices_.DeviceId(i));
      });
  }

  size_t Size() const { return perm_h_.CanRead() ? data_h_.size() : size_d_; }

  GPUSet Devices() const { return distribution_.devices_; }

  const GPUDistribution& Distribution() const { return distribution_; }

  T* DevicePointer(int device) {
    CHECK(distribution_.devices_.Contains(device));
    LazySyncDevice(device, GPUAccess::kWrite);
    return shards_.at(distribution_.devices_.Index(device)).Raw();
  }

  const T* ConstDevicePointer(int device) {
    CHECK(distribution_.devices_.Contains(device));
    LazySyncDevice(device, GPUAccess::kRead);
    return shards_.at(distribution_.devices_.Index(device)).Raw();
  }

  common::Span<T> DeviceSpan(int device) {
    GPUSet devices = distribution_.devices_;
    CHECK(devices.Contains(device));
    LazySyncDevice(device, GPUAccess::kWrite);
    return {shards_.at(devices.Index(device)).Raw(),
          static_cast<typename common::Span<T>::index_type>(DeviceSize(device))};
  }

  common::Span<const T> ConstDeviceSpan(int device) {
    GPUSet devices = distribution_.devices_;
    CHECK(devices.Contains(device));
    LazySyncDevice(device, GPUAccess::kRead);
    using SpanInd = typename common::Span<const T>::index_type;
    return {shards_.at(devices.Index(device)).Raw(),
          static_cast<SpanInd>(DeviceSize(device))};
  }

  size_t DeviceSize(int device) {
    CHECK(distribution_.devices_.Contains(device));
    LazySyncDevice(device, GPUAccess::kRead);
    return shards_.at(distribution_.devices_.Index(device)).DataSize();
  }

  size_t DeviceStart(int device) {
    CHECK(distribution_.devices_.Contains(device));
    LazySyncDevice(device, GPUAccess::kRead);
    return shards_.at(distribution_.devices_.Index(device)).Start();
  }

  thrust::device_ptr<T> tbegin(int device) {  // NOLINT
    return thrust::device_ptr<T>(DevicePointer(device));
  }

  thrust::device_ptr<const T> tcbegin(int device) {  // NOLINT
    return thrust::device_ptr<const T>(ConstDevicePointer(device));
  }

  thrust::device_ptr<T> tend(int device) {  // NOLINT
    return tbegin(device) + DeviceSize(device);
  }

  thrust::device_ptr<const T> tcend(int device) {  // NOLINT
    return tcbegin(device) + DeviceSize(device);
  }

  void ScatterFrom(thrust::device_ptr<const T> begin, thrust::device_ptr<const T> end) {
    CHECK_EQ(end - begin, Size());
    if (perm_h_.CanWrite()) {
      dh::safe_cuda(hipMemcpy(data_h_.data(), begin.get(),
                               (end - begin) * sizeof(T),
                               hipMemcpyDeviceToHost));
    } else {
    dh::ExecuteIndexShards(&shards_, [&](int idx, DeviceShard& shard) {
        shard.ScatterFrom(begin.get());
      });
    }
  }

  void GatherTo(thrust::device_ptr<T> begin, thrust::device_ptr<T> end) {
    CHECK_EQ(end - begin, Size());
    if (perm_h_.CanWrite()) {
      dh::safe_cuda(hipMemcpy(begin.get(), data_h_.data(),
                               data_h_.size() * sizeof(T),
                               hipMemcpyHostToDevice));
    } else {
      dh::ExecuteIndexShards(&shards_, [&](int idx, DeviceShard& shard) { shard.GatherTo(begin); });
    }
  }

  void Fill(T v) {  // NOLINT
    if (perm_h_.CanWrite()) {
      std::fill(data_h_.begin(), data_h_.end(), v);
    } else {
      dh::ExecuteIndexShards(&shards_, [&](int idx, DeviceShard& shard) { shard.Fill(v); });
    }
  }

  void Copy(HostDeviceVectorImpl<T>* other) {
    CHECK_EQ(Size(), other->Size());
    // Data is on host.
    if (perm_h_.CanWrite() && other->perm_h_.CanWrite()) {
      std::copy(other->data_h_.begin(), other->data_h_.end(), data_h_.begin());
      return;
    }
    // Data is on device;
    if (distribution_ != other->distribution_) {
      distribution_ = GPUDistribution();
      Shard(other->Distribution());
      size_d_ = other->size_d_;
    }
    dh::ExecuteIndexShards(&shards_, [&](int i, DeviceShard& shard) {
        shard.Copy(&other->shards_.at(i));
      });
  }

  void Copy(const std::vector<T>& other) {
    CHECK_EQ(Size(), other.size());
    if (perm_h_.CanWrite()) {
      std::copy(other.begin(), other.end(), data_h_.begin());
    } else {
      dh::ExecuteIndexShards(&shards_, [&](int idx, DeviceShard& shard) {
          shard.ScatterFrom(other.data());
        });
    }
  }

  void Copy(std::initializer_list<T> other) {
    CHECK_EQ(Size(), other.size());
    if (perm_h_.CanWrite()) {
      std::copy(other.begin(), other.end(), data_h_.begin());
    } else {
      dh::ExecuteIndexShards(&shards_, [&](int idx, DeviceShard& shard) {
          shard.ScatterFrom(other.begin());
        });
    }
  }

  std::vector<T>& HostVector() {
    LazySyncHost(GPUAccess::kWrite);
    return data_h_;
  }

  const std::vector<T>& ConstHostVector() {
    LazySyncHost(GPUAccess::kRead);
    return data_h_;
  }

  void Shard(const GPUDistribution& distribution) {
    if (distribution_ == distribution) { return; }
    CHECK(distribution_.IsEmpty())
        << "This: " << distribution_.Devices().Size() << ", "
        << "Others: " << distribution.Devices().Size();
    distribution_ = distribution;
    InitShards();
  }

  void Shard(GPUSet new_devices) {
    if (distribution_.Devices() == new_devices) { return; }
    Shard(GPUDistribution::Block(new_devices));
  }

  void Reshard(const GPUDistribution &distribution) {
    if (distribution_ == distribution) { return; }
    LazySyncHost(GPUAccess::kWrite);
    distribution_ = distribution;
    shards_.clear();
    InitShards();
  }

  void Resize(size_t new_size, T v) {
    if (new_size == Size()) { return; }
    if (distribution_.IsFixedSize()) {
      CHECK_EQ(new_size, distribution_.offsets_.back());
    }
    if (Size() == 0 && !distribution_.IsEmpty()) {
      // fast on-device resize
      perm_h_ = Permissions(false);
      size_d_ = new_size;
      InitShards();
      Fill(v);
    } else {
      // resize on host
      LazySyncHost(GPUAccess::kWrite);
      data_h_.resize(new_size, v);
    }
  }

  void LazySyncHost(GPUAccess access) {
    if (perm_h_.CanAccess(access)) { return; }
    if (perm_h_.CanRead()) {
      // data is present, just need to deny access to the device
      dh::ExecuteIndexShards(&shards_, [&](int idx, DeviceShard& shard) {
          shard.Perm().DenyComplementary(access);
        });
      perm_h_.Grant(access);
      return;
    }
    if (data_h_.size() != size_d_) { data_h_.resize(size_d_); }
    dh::ExecuteIndexShards(&shards_, [&](int idx, DeviceShard& shard) {
        shard.LazySyncHost(access);
      });
    perm_h_.Grant(access);
  }

  void LazySyncDevice(int device, GPUAccess access) {
    GPUSet devices = distribution_.Devices();
    CHECK(devices.Contains(device));
    shards_.at(devices.Index(device)).LazySyncDevice(access);
  }

  bool HostCanAccess(GPUAccess access) { return perm_h_.CanAccess(access); }

  bool DeviceCanAccess(int device, GPUAccess access) {
    GPUSet devices = distribution_.Devices();
    if (!devices.Contains(device)) { return false; }
    return shards_.at(devices.Index(device)).Perm().CanAccess(access);
  }

 private:
  std::vector<T> data_h_;
  Permissions perm_h_;
  // the total size of the data stored on the devices
  size_t size_d_;
  GPUDistribution distribution_;
  // protects size_d_ and perm_h_ when updated from multiple threads
  std::mutex mutex_;
  std::vector<DeviceShard> shards_;
};

template <typename T>
HostDeviceVector<T>::HostDeviceVector
(size_t size, T v, const GPUDistribution &distribution) : impl_(nullptr) {
  impl_ = new HostDeviceVectorImpl<T>(size, v, distribution);
}

template <typename T>
HostDeviceVector<T>::HostDeviceVector
(std::initializer_list<T> init, const GPUDistribution &distribution) : impl_(nullptr) {
  impl_ = new HostDeviceVectorImpl<T>(init, distribution);
}

template <typename T>
HostDeviceVector<T>::HostDeviceVector
(const std::vector<T>& init, const GPUDistribution &distribution) : impl_(nullptr) {
  impl_ = new HostDeviceVectorImpl<T>(init, distribution);
}

template <typename T>
HostDeviceVector<T>::HostDeviceVector(const HostDeviceVector<T>& other)
  : impl_(nullptr) {
  impl_ = new HostDeviceVectorImpl<T>(*other.impl_);
}

template <typename T>
HostDeviceVector<T>& HostDeviceVector<T>::operator=
(const HostDeviceVector<T>& other) {
  if (this == &other) { return *this; }

  std::unique_ptr<HostDeviceVectorImpl<T>> newImpl(new HostDeviceVectorImpl<T>(*other.impl_));
  delete impl_;
  impl_ = newImpl.release();
  return *this;
}

template <typename T>
HostDeviceVector<T>::~HostDeviceVector() {
  delete impl_;
  impl_ = nullptr;
}

template <typename T>
size_t HostDeviceVector<T>::Size() const { return impl_->Size(); }

template <typename T>
GPUSet HostDeviceVector<T>::Devices() const { return impl_->Devices(); }

template <typename T>
const GPUDistribution& HostDeviceVector<T>::Distribution() const {
  return impl_->Distribution();
}

template <typename T>
T* HostDeviceVector<T>::DevicePointer(int device) {
  return impl_->DevicePointer(device);
}

template <typename T>
const T* HostDeviceVector<T>::ConstDevicePointer(int device) const {
  return impl_->ConstDevicePointer(device);
}

template <typename T>
common::Span<T> HostDeviceVector<T>::DeviceSpan(int device) {
  return impl_->DeviceSpan(device);
}

template <typename T>
common::Span<const T> HostDeviceVector<T>::ConstDeviceSpan(int device) const {
  return impl_->ConstDeviceSpan(device);
}

template <typename T>
size_t HostDeviceVector<T>::DeviceStart(int device) const {
  return impl_->DeviceStart(device);
}

template <typename T>
size_t HostDeviceVector<T>::DeviceSize(int device) const {
  return impl_->DeviceSize(device);
}

template <typename T>
thrust::device_ptr<T> HostDeviceVector<T>::tbegin(int device) {  // NOLINT
  return impl_->tbegin(device);
}

template <typename T>
thrust::device_ptr<const T> HostDeviceVector<T>::tcbegin(int device) const {  // NOLINT
  return impl_->tcbegin(device);
}

template <typename T>
thrust::device_ptr<T> HostDeviceVector<T>::tend(int device) {  // NOLINT
  return impl_->tend(device);
}

template <typename T>
thrust::device_ptr<const T> HostDeviceVector<T>::tcend(int device) const {  // NOLINT
  return impl_->tcend(device);
}

template <typename T>
void HostDeviceVector<T>::ScatterFrom
(thrust::device_ptr<const T> begin, thrust::device_ptr<const T> end) {
  impl_->ScatterFrom(begin, end);
}

template <typename T>
void HostDeviceVector<T>::GatherTo
(thrust::device_ptr<T> begin, thrust::device_ptr<T> end) const {
  impl_->GatherTo(begin, end);
}

template <typename T>
void HostDeviceVector<T>::Fill(T v) {
  impl_->Fill(v);
}

template <typename T>
void HostDeviceVector<T>::Copy(const HostDeviceVector<T>& other) {
  impl_->Copy(other.impl_);
}

template <typename T>
void HostDeviceVector<T>::Copy(const std::vector<T>& other) {
  impl_->Copy(other);
}

template <typename T>
void HostDeviceVector<T>::Copy(std::initializer_list<T> other) {
  impl_->Copy(other);
}

template <typename T>
std::vector<T>& HostDeviceVector<T>::HostVector() { return impl_->HostVector(); }

template <typename T>
const std::vector<T>& HostDeviceVector<T>::ConstHostVector() const {
  return impl_->ConstHostVector();
}

template <typename T>
bool HostDeviceVector<T>::HostCanAccess(GPUAccess access) const {
  return impl_->HostCanAccess(access);
}

template <typename T>
bool HostDeviceVector<T>::DeviceCanAccess(int device, GPUAccess access) const {
  return impl_->DeviceCanAccess(device, access);
}

template <typename T>
void HostDeviceVector<T>::Shard(GPUSet new_devices) const {
  impl_->Shard(new_devices);
}

template <typename T>
void HostDeviceVector<T>::Shard(const GPUDistribution &distribution) const {
  impl_->Shard(distribution);
}

template <typename T>
void HostDeviceVector<T>::Reshard(const GPUDistribution &distribution) {
  impl_->Reshard(distribution);
}

template <typename T>
void HostDeviceVector<T>::Resize(size_t new_size, T v) {
  impl_->Resize(new_size, v);
}

// explicit instantiations are required, as HostDeviceVector isn't header-only
template class HostDeviceVector<bst_float>;
template class HostDeviceVector<GradientPair>;
template class HostDeviceVector<int>;
template class HostDeviceVector<Entry>;
template class HostDeviceVector<size_t>;

}  // namespace xgboost
