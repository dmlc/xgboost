/*!
 * Copyright 2017 XGBoost contributors
 */

#include "./host_device_vector.h"
#include "./device_helpers.cuh"

namespace xgboost {

template <typename T>
struct HostDeviceVectorImpl {
  HostDeviceVectorImpl(size_t size, T v, int device)
    : device_(device), on_d_(device >= 0) {
    if (on_d_) {
      dh::safe_cuda(hipSetDevice(device_));
      data_d_.resize(size, v);
    } else {
      data_h_.resize(size, v);
    }
  }
  // Init can be std::vector<T> or std::initializer_list<T>
  template <class Init>
  HostDeviceVectorImpl(const Init& init, int device)
    : device_(device), on_d_(device >= 0) {
    if (on_d_) {
      dh::safe_cuda(hipSetDevice(device_));
      data_d_.resize(init.size());
      thrust::copy(init.begin(), init.end(), data_d_.begin());
    } else {
      data_h_ = init;
    }
  }
  HostDeviceVectorImpl(const HostDeviceVectorImpl<T>&) = delete;
  HostDeviceVectorImpl(HostDeviceVectorImpl<T>&&) = delete;
  void operator=(const HostDeviceVectorImpl<T>&) = delete;
  void operator=(HostDeviceVectorImpl<T>&&) = delete;

  size_t size() const { return on_d_ ? data_d_.size() : data_h_.size(); }

  int device() const { return device_; }

  T* ptr_d(int device) {
    lazy_sync_device(device);
    return data_d_.data().get();
  }
  thrust::device_ptr<T> tbegin(int device) {
    return thrust::device_ptr<T>(ptr_d(device));
  }
  thrust::device_ptr<T> tend(int device) {
    auto begin = tbegin(device);
    return begin + size();
  }
  std::vector<T>& data_h() {
    lazy_sync_host();
    return data_h_;
  }
  void resize(size_t new_size, T v, int new_device) {
    if (new_size == this->size() && new_device == device_)
      return;
    if (new_device != -1)
      device_ = new_device;
    // if !on_d_, but the data size is 0 and the device is set,
    // resize the data on device instead
    if (!on_d_ && (data_h_.size() > 0 || device_ == -1)) {
      data_h_.resize(new_size, v);
    } else {
      dh::safe_cuda(hipSetDevice(device_));
      data_d_.resize(new_size, v);
      on_d_ = true;
    }
  }

  void lazy_sync_host() {
    if (!on_d_)
      return;
    if (data_h_.size() != this->size())
      data_h_.resize(this->size());
    dh::safe_cuda(hipSetDevice(device_));
    thrust::copy(data_d_.begin(), data_d_.end(), data_h_.begin());
    on_d_ = false;
  }

  void lazy_sync_device(int device) {
    if (on_d_)
      return;
    if (device != device_) {
      CHECK_EQ(device_, -1);
      device_ = device;
    }
    if (data_d_.size() != this->size()) {
      dh::safe_cuda(hipSetDevice(device_));
      data_d_.resize(this->size());
    }
    dh::safe_cuda(hipSetDevice(device_));
    thrust::copy(data_h_.begin(), data_h_.end(), data_d_.begin());
    on_d_ = true;
  }

  std::vector<T> data_h_;
  thrust::device_vector<T> data_d_;
  // true if there is an up-to-date copy of data on device, false otherwise
  bool on_d_;
  int device_;
};

template <typename T>
HostDeviceVector<T>::HostDeviceVector(size_t size, T v, int device) : impl_(nullptr) {
  impl_ = new HostDeviceVectorImpl<T>(size, v, device);
}

template <typename T>
HostDeviceVector<T>::HostDeviceVector(std::initializer_list<T> init, int device)
  : impl_(nullptr) {
  impl_ = new HostDeviceVectorImpl<T>(init, device);
}

template <typename T>
HostDeviceVector<T>::HostDeviceVector(const std::vector<T>& init, int device)
  : impl_(nullptr) {
  impl_ = new HostDeviceVectorImpl<T>(init, device);
}

template <typename T>
HostDeviceVector<T>::~HostDeviceVector() {
  HostDeviceVectorImpl<T>* tmp = impl_;
  impl_ = nullptr;
  delete tmp;
}

template <typename T>
size_t HostDeviceVector<T>::size() const { return impl_->size(); }

template <typename T>
int HostDeviceVector<T>::device() const { return impl_->device(); }

template <typename T>
T* HostDeviceVector<T>::ptr_d(int device) { return impl_->ptr_d(device); }

template <typename T>
thrust::device_ptr<T> HostDeviceVector<T>::tbegin(int device) {
  return impl_->tbegin(device);
}

template <typename T>
thrust::device_ptr<T> HostDeviceVector<T>::tend(int device) {
  return impl_->tend(device);
}

template <typename T>
std::vector<T>& HostDeviceVector<T>::data_h() { return impl_->data_h(); }

template <typename T>
void HostDeviceVector<T>::resize(size_t new_size, T v, int new_device) {
  impl_->resize(new_size, v, new_device);
}

// explicit instantiations are required, as HostDeviceVector isn't header-only
template class HostDeviceVector<bst_float>;
template class HostDeviceVector<bst_gpair>;

}  // namespace xgboost
