#include "hip/hip_runtime.h"
/*!
 * Copyright 2020 by XGBoost Contributors
 */
#include <thrust/unique.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/binary_search.h>
#include <thrust/transform_scan.h>
#include <thrust/execution_policy.h>

#include <memory>
#include <utility>

#include "xgboost/span.h"
#include "quantile.h"
#include "quantile.cuh"
#include "hist_util.h"
#include "device_helpers.cuh"
#include "categorical.h"
#include "common.h"

namespace xgboost {
namespace common {

using WQSketch = HostSketchContainer::WQSketch;
using SketchEntry = WQSketch::Entry;

// Algorithm 4 in XGBoost's paper, using binary search to find i.
template <typename EntryIter>
__device__ SketchEntry BinarySearchQuery(EntryIter beg, EntryIter end, float rank) {
  assert(end - beg >= 2);
  rank *= 2;
  auto front = *beg;
  if (rank < front.rmin + front.rmax) {
    return *beg;
  }
  auto back = *(end - 1);
  if (rank >= back.rmin + back.rmax) {
    return back;
  }

  auto search_begin = dh::MakeTransformIterator<float>(
      beg, [=] __device__(SketchEntry const &entry) {
        return entry.rmin + entry.rmax;
      });
  auto search_end = search_begin + (end - beg);
  auto i =
      thrust::upper_bound(thrust::seq, search_begin + 1, search_end - 1, rank) -
      search_begin - 1;
  if (rank < (*(beg + i)).RMinNext() + (*(beg + i + 1)).RMaxPrev()) {
    return *(beg + i);
  } else {
    return *(beg + i + 1);
  }
}

template <typename InEntry, typename ToSketchEntry>
void PruneImpl(int device,
               common::Span<SketchContainer::OffsetT const> cuts_ptr,
               Span<InEntry const> sorted_data,
               Span<size_t const> columns_ptr_in,  // could be ptr for data or cuts
               Span<FeatureType const> feature_types,
               Span<SketchEntry> out_cuts,
               ToSketchEntry to_sketch_entry) {
  dh::LaunchN(device, out_cuts.size(), [=] __device__(size_t idx) {
    size_t column_id = dh::SegmentId(cuts_ptr, idx);
    auto out_column = out_cuts.subspan(
        cuts_ptr[column_id], cuts_ptr[column_id + 1] - cuts_ptr[column_id]);
    auto in_column = sorted_data.subspan(columns_ptr_in[column_id],
                                         columns_ptr_in[column_id + 1] -
                                             columns_ptr_in[column_id]);
    auto to = cuts_ptr[column_id + 1] - cuts_ptr[column_id];
    idx -= cuts_ptr[column_id];
    auto front = to_sketch_entry(0ul, in_column, column_id);
    auto back = to_sketch_entry(in_column.size() - 1, in_column, column_id);

    auto is_cat = IsCat(feature_types, column_id);
    if (in_column.size() <= to || is_cat) {
      // cut idx equals sample idx
      out_column[idx] = to_sketch_entry(idx, in_column, column_id);
      return;
    }
    // 1 thread for each output.  See A.4 for detail.
    auto d_out = out_column;
    if (idx == 0) {
      d_out.front() = front;
      return;
    }
    if (idx == to - 1) {
      d_out.back() = back;
      return;
    }

    float w = back.rmin - front.rmax;
    assert(w != 0);
    auto budget = static_cast<float>(d_out.size());
    assert(budget != 0);
    auto q = ((static_cast<float>(idx) * w) / (static_cast<float>(to) - 1.0f) + front.rmax);
    auto it = dh::MakeTransformIterator<SketchEntry>(
        thrust::make_counting_iterator(0ul), [=] __device__(size_t idx) {
          auto e = to_sketch_entry(idx, in_column, column_id);
          return e;
        });
    d_out[idx] = BinarySearchQuery(it, it + in_column.size(), q);
  });
}

template <typename T, typename U>
void CopyTo(Span<T> out, Span<U> src) {
  CHECK_EQ(out.size(), src.size());
  static_assert(std::is_same<std::remove_cv_t<T>, std::remove_cv_t<T>>::value, "");
  dh::safe_cuda(hipMemcpyAsync(out.data(), src.data(),
                                out.size_bytes(),
                                hipMemcpyDefault));
}

// Compute the merge path.
common::Span<thrust::tuple<uint64_t, uint64_t>> MergePath(
    Span<SketchEntry const> const &d_x, Span<bst_row_t const> const &x_ptr,
    Span<SketchEntry const> const &d_y, Span<bst_row_t const> const &y_ptr,
    Span<SketchEntry> out, Span<bst_row_t> out_ptr) {
  auto x_merge_key_it = thrust::make_zip_iterator(thrust::make_tuple(
      dh::MakeTransformIterator<bst_row_t>(
          thrust::make_counting_iterator(0ul),
          [=] __device__(size_t idx) { return dh::SegmentId(x_ptr, idx); }),
      d_x.data()));
  auto y_merge_key_it = thrust::make_zip_iterator(thrust::make_tuple(
      dh::MakeTransformIterator<bst_row_t>(
          thrust::make_counting_iterator(0ul),
          [=] __device__(size_t idx) { return dh::SegmentId(y_ptr, idx); }),
      d_y.data()));

  using Tuple = thrust::tuple<uint64_t, uint64_t>;

  thrust::constant_iterator<uint64_t> a_ind_iter(0ul);
  thrust::constant_iterator<uint64_t> b_ind_iter(1ul);

  auto place_holder = thrust::make_constant_iterator<uint64_t>(0u);
  auto x_merge_val_it =
      thrust::make_zip_iterator(thrust::make_tuple(a_ind_iter, place_holder));
  auto y_merge_val_it =
      thrust::make_zip_iterator(thrust::make_tuple(b_ind_iter, place_holder));

  dh::XGBCachingDeviceAllocator<Tuple> alloc;
  static_assert(sizeof(Tuple) == sizeof(SketchEntry), "");
  // We reuse the memory for storing merge path.
  common::Span<Tuple> merge_path{reinterpret_cast<Tuple *>(out.data()), out.size()};
  // Determine the merge path, 0 if element is from x, 1 if it's from y.
  thrust::merge_by_key(
      thrust::cuda::par(alloc), x_merge_key_it, x_merge_key_it + d_x.size(),
      y_merge_key_it, y_merge_key_it + d_y.size(), x_merge_val_it,
      y_merge_val_it, thrust::make_discard_iterator(), merge_path.data(),
      [=] __device__(auto const &l, auto const &r) -> bool {
        auto l_column_id = thrust::get<0>(l);
        auto r_column_id = thrust::get<0>(r);
        if (l_column_id == r_column_id) {
          return thrust::get<1>(l).value < thrust::get<1>(r).value;
        }
        return l_column_id < r_column_id;
      });

  // Compute output ptr
  auto transform_it =
      thrust::make_zip_iterator(thrust::make_tuple(x_ptr.data(), y_ptr.data()));
  thrust::transform(
      thrust::cuda::par(alloc), transform_it, transform_it + x_ptr.size(),
      out_ptr.data(),
      [] __device__(auto const& t) { return thrust::get<0>(t) + thrust::get<1>(t); });

  // 0^th is the indicator, 1^th is placeholder
  auto get_ind = []XGBOOST_DEVICE(Tuple const& t) { return thrust::get<0>(t); };
  // 0^th is the counter for x, 1^th for y.
  auto get_x =   []XGBOOST_DEVICE(Tuple const &t) { return thrust::get<0>(t); };
  auto get_y =   []XGBOOST_DEVICE(Tuple const &t) { return thrust::get<1>(t); };

  auto scan_key_it = dh::MakeTransformIterator<size_t>(
      thrust::make_counting_iterator(0ul),
      [=] __device__(size_t idx) { return dh::SegmentId(out_ptr, idx); });

  auto scan_val_it = dh::MakeTransformIterator<Tuple>(
      merge_path.data(), [=] __device__(Tuple const &t) -> Tuple {
        auto ind = get_ind(t);  // == 0 if element is from x
        // x_counter, y_counter
        return thrust::make_tuple<uint64_t, uint64_t>(!ind, ind);
      });

  // Compute the index for both x and y (which of the element in a and b are used in each
  // comparison) by scaning the binary merge path.  Take output [(x_0, y_0), (x_0, y_1),
  // ...] as an example, the comparison between (x_0, y_0) adds 1 step in the merge path.
  // Asumming y_0 is less than x_0 so this step is torward the end of y.  After the
  // comparison, index of y is incremented by 1 from y_0 to y_1, and at the same time, y_0
  // is landed into output as the first element in merge result.  The scan result is the
  // subscript of x and y.
  thrust::exclusive_scan_by_key(
      thrust::cuda::par(alloc), scan_key_it, scan_key_it + merge_path.size(),
      scan_val_it, merge_path.data(),
      thrust::make_tuple<uint64_t, uint64_t>(0ul, 0ul),
      thrust::equal_to<size_t>{},
      [=] __device__(Tuple const &l, Tuple const &r) -> Tuple {
        return thrust::make_tuple(get_x(l) + get_x(r), get_y(l) + get_y(r));
      });

  return merge_path;
}

// Merge d_x and d_y into out.  Because the final output depends on predicate (which
// summary does the output element come from) result by definition of merged rank.  So we
// run it in 2 passes to obtain the merge path and then customize the standard merge
// algorithm.
void MergeImpl(int32_t device, Span<SketchEntry const> const &d_x,
               Span<bst_row_t const> const &x_ptr,
               Span<SketchEntry const> const &d_y,
               Span<bst_row_t const> const &y_ptr,
               Span<SketchEntry> out,
               Span<bst_row_t> out_ptr) {
  dh::safe_cuda(hipSetDevice(device));
  CHECK_EQ(d_x.size() + d_y.size(), out.size());
  CHECK_EQ(x_ptr.size(), out_ptr.size());
  CHECK_EQ(y_ptr.size(), out_ptr.size());

  auto d_merge_path = MergePath(d_x, x_ptr, d_y, y_ptr, out, out_ptr);
  auto d_out = out;

  dh::LaunchN(device, d_out.size(), [=] __device__(size_t idx) {
    auto column_id = dh::SegmentId(out_ptr, idx);
    idx -= out_ptr[column_id];

    auto d_x_column =
        d_x.subspan(x_ptr[column_id], x_ptr[column_id + 1] - x_ptr[column_id]);
    auto d_y_column =
        d_y.subspan(y_ptr[column_id], y_ptr[column_id + 1] - y_ptr[column_id]);
    auto d_out_column = d_out.subspan(
        out_ptr[column_id], out_ptr[column_id + 1] - out_ptr[column_id]);
    auto d_path_column = d_merge_path.subspan(
        out_ptr[column_id], out_ptr[column_id + 1] - out_ptr[column_id]);

    uint64_t a_ind, b_ind;
    thrust::tie(a_ind, b_ind) = d_path_column[idx];

    // Handle empty column.  If both columns are empty, we should not get this column_id
    // as result of binary search.
    assert((d_x_column.size() != 0) || (d_y_column.size() != 0));
    if (d_x_column.size() == 0) {
      d_out_column[idx] = d_y_column[b_ind];
      return;
    }
    if (d_y_column.size() == 0) {
      d_out_column[idx] = d_x_column[a_ind];
      return;
    }

    // Handle trailing elements.
    assert(a_ind <= d_x_column.size());
    if (a_ind == d_x_column.size()) {
      // Trailing elements are from y because there's no more x to land.
      auto y_elem = d_y_column[b_ind];
      d_out_column[idx] = SketchEntry(y_elem.rmin + d_x_column.back().RMinNext(),
                                      y_elem.rmax + d_x_column.back().rmax,
                                      y_elem.wmin, y_elem.value);
      return;
    }
    auto x_elem = d_x_column[a_ind];
    assert(b_ind <= d_y_column.size());
    if (b_ind == d_y_column.size()) {
      d_out_column[idx] = SketchEntry(x_elem.rmin + d_y_column.back().RMinNext(),
                                      x_elem.rmax + d_y_column.back().rmax,
                                      x_elem.wmin, x_elem.value);
      return;
    }
    auto y_elem = d_y_column[b_ind];

    /* Merge procedure.  See A.3 merge operation eq (26) ~ (28).  The trick to interpret
       it is rewriting the symbols on both side of equality.  Take eq (26) as an example:
       Expand it according to definition of extended rank then rewrite it into:

       If $k_i$ is the $i$ element in output and \textbf{comes from $D_1$}:

         r_\bar{D}(k_i) = r_{\bar{D_1}}(k_i) + w_{\bar{{D_1}}}(k_i) +
                                          [r_{\bar{D_2}}(x_i) + w_{\bar{D_2}}(x_i)]

       Where $x_i$ is the largest element in $D_2$ that's less than $k_i$.  $k_i$ can be
       used in $D_1$ as it's since $k_i \in D_1$.  Other 2 equations can be applied
       similarly with $k_i$ comes from different $D$.  just use different symbol on
       different source of summary.
    */
    assert(idx < d_out_column.size());
    if (x_elem.value == y_elem.value) {
      d_out_column[idx] =
          SketchEntry{x_elem.rmin + y_elem.rmin, x_elem.rmax + y_elem.rmax,
                      x_elem.wmin + y_elem.wmin, x_elem.value};
    } else if (x_elem.value < y_elem.value) {
      // elem from x is landed. yprev_min is the element in D_2 that's 1 rank less than
      // x_elem if we put x_elem in D_2.
      float yprev_min = b_ind == 0 ? 0.0f : d_y_column[b_ind - 1].RMinNext();
      // rmin should be equal to x_elem.rmin + x_elem.wmin + yprev_min.  But for
      // implementation, the weight is stored in a separated field and we compute the
      // extended definition on the fly when needed.
      d_out_column[idx] =
          SketchEntry{x_elem.rmin + yprev_min, x_elem.rmax + y_elem.RMaxPrev(),
                      x_elem.wmin, x_elem.value};
    } else {
      // elem from y is landed.
      float xprev_min = a_ind == 0 ? 0.0f : d_x_column[a_ind - 1].RMinNext();
      d_out_column[idx] =
          SketchEntry{xprev_min + y_elem.rmin, x_elem.RMaxPrev() + y_elem.rmax,
                      y_elem.wmin, y_elem.value};
    }
  });
}

void SketchContainer::Push(Span<Entry const> entries, Span<size_t> columns_ptr,
                           common::Span<OffsetT> cuts_ptr,
                           size_t total_cuts, Span<float> weights) {
  Span<SketchEntry> out;
  dh::device_vector<SketchEntry> cuts;
  bool first_window = this->Current().empty();
  if (!first_window) {
    cuts.resize(total_cuts);
    out = dh::ToSpan(cuts);
  } else {
    this->Current().resize(total_cuts);
    out = dh::ToSpan(this->Current());
  }
  auto ft = this->feature_types_.ConstDeviceSpan();
  if (weights.empty()) {
    auto to_sketch_entry = [] __device__(size_t sample_idx,
                                         Span<Entry const> const &column,
                                         size_t) {
      float rmin = sample_idx;
      float rmax = sample_idx + 1;
      return SketchEntry{rmin, rmax, 1, column[sample_idx].fvalue};
    }; // NOLINT
    PruneImpl<Entry>(device_, cuts_ptr, entries, columns_ptr, ft, out,
                     to_sketch_entry);
  } else {
    auto to_sketch_entry = [weights, columns_ptr] __device__(
                               size_t sample_idx,
                               Span<Entry const> const &column,
                               size_t column_id) {
      Span<float const> column_weights_scan =
          weights.subspan(columns_ptr[column_id], column.size());
      float rmin = sample_idx > 0 ? column_weights_scan[sample_idx - 1] : 0.0f;
      float rmax = column_weights_scan[sample_idx];
      float wmin = rmax - rmin;
      wmin = wmin < 0 ? kRtEps : wmin;  // GPU scan can generate floating error.
      return SketchEntry{rmin, rmax, wmin, column[sample_idx].fvalue};
    }; // NOLINT
    PruneImpl<Entry>(device_, cuts_ptr, entries, columns_ptr, ft, out,
                     to_sketch_entry);
  }
  auto n_uniques = this->ScanInput(out, cuts_ptr);

  if (!first_window) {
    CHECK_EQ(this->columns_ptr_.Size(), cuts_ptr.size());
    out = out.subspan(0, n_uniques);
    this->Merge(cuts_ptr, out);
    this->FixError();
  } else {
    this->Current().resize(n_uniques);
    this->columns_ptr_.SetDevice(device_);
    this->columns_ptr_.Resize(cuts_ptr.size());

    auto d_cuts_ptr = this->columns_ptr_.DeviceSpan();
    CopyTo(d_cuts_ptr, cuts_ptr);
  }
}

size_t SketchContainer::ScanInput(Span<SketchEntry> entries, Span<OffsetT> d_columns_ptr_in) {
  /* There are 2 types of duplication.  First is duplicated feature values, which comes
   * from user input data.  Second is duplicated sketching entries, which is generated by
   * prunning or merging. We preserve the first type and remove the second type.
   */
  timer_.Start(__func__);
  dh::safe_cuda(hipSetDevice(device_));
  CHECK_EQ(d_columns_ptr_in.size(), num_columns_ + 1);
  dh::XGBCachingDeviceAllocator<char> alloc;

  auto key_it = dh::MakeTransformIterator<size_t>(
      thrust::make_reverse_iterator(thrust::make_counting_iterator(entries.size())),
      [=] __device__(size_t idx) {
        return dh::SegmentId(d_columns_ptr_in, idx);
      });
  // Reverse scan to accumulate weights into first duplicated element on left.
  auto val_it = thrust::make_reverse_iterator(dh::tend(entries));
  thrust::inclusive_scan_by_key(
      thrust::cuda::par(alloc), key_it, key_it + entries.size(),
      val_it, val_it,
      thrust::equal_to<size_t>{},
      [] __device__(SketchEntry const &r, SketchEntry const &l) {
        // Only accumulate for the first type of duplication.
        if (l.value - r.value == 0 && l.rmin - r.rmin != 0) {
          auto w = l.wmin + r.wmin;
          SketchEntry v{l.rmin, l.rmin + w, w, l.value};
          return v;
        }
        return l;
      });

  auto d_columns_ptr_out = columns_ptr_b_.DeviceSpan();
  // thrust unique_by_key preserves the first element.
  auto n_uniques = dh::SegmentedUnique(
      d_columns_ptr_in.data(),
      d_columns_ptr_in.data() + d_columns_ptr_in.size(), entries.data(),
      entries.data() + entries.size(), d_columns_ptr_out.data(), entries.data(),
      detail::SketchUnique{});
  CopyTo(d_columns_ptr_in, d_columns_ptr_out);

  timer_.Stop(__func__);
  return n_uniques;
}

size_t SketchContainer::Unique() {
  timer_.Start(__func__);
  dh::safe_cuda(hipSetDevice(device_));
  this->columns_ptr_.SetDevice(device_);
  Span<OffsetT> d_column_scan = this->columns_ptr_.DeviceSpan();
  CHECK_EQ(d_column_scan.size(), num_columns_ + 1);
  Span<SketchEntry> entries = dh::ToSpan(this->Current());
  HostDeviceVector<OffsetT> scan_out(d_column_scan.size());
  scan_out.SetDevice(device_);
  auto d_scan_out = scan_out.DeviceSpan();

  d_column_scan = this->columns_ptr_.DeviceSpan();
  size_t n_uniques = dh::SegmentedUnique(
      d_column_scan.data(), d_column_scan.data() + d_column_scan.size(),
      entries.data(), entries.data() + entries.size(), scan_out.DevicePointer(),
      entries.data(),
      detail::SketchUnique{});
  this->columns_ptr_.Copy(scan_out);
  CHECK(!this->columns_ptr_.HostCanRead());

  this->Current().resize(n_uniques);
  timer_.Stop(__func__);
  return n_uniques;
}

void SketchContainer::Prune(size_t to) {
  timer_.Start(__func__);
  dh::safe_cuda(hipSetDevice(device_));

  OffsetT to_total = 0;
  auto& h_columns_ptr = columns_ptr_b_.HostVector();
  h_columns_ptr[0] = to_total;
  auto const& h_feature_types = feature_types_.ConstHostSpan();
  for (bst_feature_t i = 0; i < num_columns_; ++i) {
    size_t length = this->Column(i).size();
    length = std::min(length, to);
    if (IsCat(h_feature_types, i)) {
      length = this->Column(i).size();
    }
    to_total += length;
    h_columns_ptr[i+1] = to_total;
  }
  this->Other().resize(to_total);

  auto d_columns_ptr_in = this->columns_ptr_.ConstDeviceSpan();
  auto d_columns_ptr_out = columns_ptr_b_.ConstDeviceSpan();
  auto out = dh::ToSpan(this->Other());
  auto in = dh::ToSpan(this->Current());
  auto no_op = [] __device__(size_t sample_idx,
                             Span<SketchEntry const> const &entries,
                             size_t) { return entries[sample_idx]; }; // NOLINT
  auto ft = this->feature_types_.ConstDeviceSpan();
  PruneImpl<SketchEntry>(device_, d_columns_ptr_out, in, d_columns_ptr_in, ft,
                         out, no_op);
  this->columns_ptr_.Copy(columns_ptr_b_);
  this->Alternate();

  this->Unique();
  timer_.Stop(__func__);
}

void SketchContainer::Merge(Span<OffsetT const> d_that_columns_ptr,
                            Span<SketchEntry const> that) {
  dh::safe_cuda(hipSetDevice(device_));
  timer_.Start(__func__);
  if (this->Current().size() == 0) {
    CHECK_EQ(this->columns_ptr_.HostVector().back(), 0);
    CHECK_EQ(this->columns_ptr_.HostVector().size(), d_that_columns_ptr.size());
    CHECK_EQ(columns_ptr_.Size(), num_columns_ + 1);
    thrust::copy(thrust::device, d_that_columns_ptr.data(),
                 d_that_columns_ptr.data() + d_that_columns_ptr.size(),
                 this->columns_ptr_.DevicePointer());
    auto total = this->columns_ptr_.HostVector().back();
    this->Current().resize(total);
    CopyTo(dh::ToSpan(this->Current()), that);
    timer_.Stop(__func__);
    return;
  }

  this->Other().resize(this->Current().size() + that.size());
  CHECK_EQ(d_that_columns_ptr.size(), this->columns_ptr_.Size());

  MergeImpl(device_, this->Data(), this->ColumnsPtr(),
            that, d_that_columns_ptr,
            dh::ToSpan(this->Other()), columns_ptr_b_.DeviceSpan());
  this->columns_ptr_.Copy(columns_ptr_b_);
  CHECK_EQ(this->columns_ptr_.Size(), num_columns_ + 1);
  this->Alternate();

  timer_.Stop(__func__);
}

void SketchContainer::FixError() {
  dh::safe_cuda(hipSetDevice(device_));
  auto d_columns_ptr = this->columns_ptr_.ConstDeviceSpan();
  auto in = dh::ToSpan(this->Current());
  dh::LaunchN(device_, in.size(), [=] __device__(size_t idx) {
    auto column_id = dh::SegmentId(d_columns_ptr, idx);
    auto in_column = in.subspan(d_columns_ptr[column_id],
                                d_columns_ptr[column_id + 1] -
                                    d_columns_ptr[column_id]);
    idx -= d_columns_ptr[column_id];
    float prev_rmin = idx == 0 ? 0.0f : in_column[idx-1].rmin;
    if (in_column[idx].rmin < prev_rmin) {
      in_column[idx].rmin = prev_rmin;
    }
    float prev_rmax = idx == 0 ? 0.0f : in_column[idx-1].rmax;
    if (in_column[idx].rmax < prev_rmax) {
      in_column[idx].rmax = prev_rmax;
    }
    float rmin_next = in_column[idx].RMinNext();
    if (in_column[idx].rmax < rmin_next) {
      in_column[idx].rmax = rmin_next;
    }
  });
}

void SketchContainer::AllReduce() {
  dh::safe_cuda(hipSetDevice(device_));
  auto world = rabit::GetWorldSize();
  if (world == 1) {
    return;
  }

  timer_.Start(__func__);
  if (!reducer_) {
    reducer_ = std::make_unique<dh::AllReducer>();
    reducer_->Init(device_);
  }
  // Reduce the overhead on syncing.
  size_t global_sum_rows = num_rows_;
  rabit::Allreduce<rabit::op::Sum>(&global_sum_rows, 1);
  size_t intermediate_num_cuts =
      std::min(global_sum_rows, static_cast<size_t>(num_bins_ * kFactor));
  this->Prune(intermediate_num_cuts);

  auto d_columns_ptr = this->columns_ptr_.ConstDeviceSpan();
  CHECK_EQ(d_columns_ptr.size(), num_columns_ + 1);
  size_t n = d_columns_ptr.size();
  rabit::Allreduce<rabit::op::Max>(&n, 1);
  CHECK_EQ(n, d_columns_ptr.size()) << "Number of columns differs across workers";

  // Get the columns ptr from all workers
  dh::device_vector<SketchContainer::OffsetT> gathered_ptrs;
  gathered_ptrs.resize(d_columns_ptr.size() * world, 0);
  size_t rank = rabit::GetRank();
  auto offset = rank * d_columns_ptr.size();
  thrust::copy(thrust::device, d_columns_ptr.data(), d_columns_ptr.data() + d_columns_ptr.size(),
               gathered_ptrs.begin() + offset);
  reducer_->AllReduceSum(gathered_ptrs.data().get(), gathered_ptrs.data().get(),
                         gathered_ptrs.size());

  // Get the data from all workers.
  std::vector<size_t> recv_lengths;
  dh::caching_device_vector<char> recvbuf;
  reducer_->AllGather(this->Current().data().get(),
                      dh::ToSpan(this->Current()).size_bytes(), &recv_lengths,
                      &recvbuf);
  reducer_->Synchronize();

  // Segment the received data.
  auto s_recvbuf = dh::ToSpan(recvbuf);
  std::vector<Span<SketchEntry>> allworkers;
  offset = 0;
  for (int32_t i = 0; i < world; ++i) {
    size_t length_as_bytes = recv_lengths.at(i);
    auto raw = s_recvbuf.subspan(offset, length_as_bytes);
    auto sketch = Span<SketchEntry>(reinterpret_cast<SketchEntry *>(raw.data()),
                                    length_as_bytes / sizeof(SketchEntry));
    allworkers.emplace_back(sketch);
    offset += length_as_bytes;
  }

  // Merge them into a new sketch.
  SketchContainer new_sketch(this->feature_types_, num_bins_,
                             this->num_columns_, global_sum_rows,
                             this->device_);
  for (size_t i = 0; i < allworkers.size(); ++i) {
    auto worker = allworkers[i];
    auto worker_ptr =
        dh::ToSpan(gathered_ptrs)
            .subspan(i * d_columns_ptr.size(), d_columns_ptr.size());
    new_sketch.Merge(worker_ptr, worker);
    new_sketch.FixError();
  }

  *this = std::move(new_sketch);
  timer_.Stop(__func__);
}

void SketchContainer::MakeCuts(HistogramCuts* p_cuts) {
  timer_.Start(__func__);
  dh::safe_cuda(hipSetDevice(device_));
  p_cuts->min_vals_.Resize(num_columns_);

  // Sync between workers.
  this->AllReduce();

  // Prune to final number of bins.
  this->Prune(num_bins_ + 1);
  this->FixError();

  // Set up inputs
  auto d_in_columns_ptr = this->columns_ptr_.ConstDeviceSpan();

  p_cuts->min_vals_.SetDevice(device_);
  auto d_min_values = p_cuts->min_vals_.DeviceSpan();
  auto in_cut_values = dh::ToSpan(this->Current());

  // Set up output ptr
  p_cuts->cut_ptrs_.SetDevice(device_);
  auto& h_out_columns_ptr = p_cuts->cut_ptrs_.HostVector();
  h_out_columns_ptr.clear();
  h_out_columns_ptr.push_back(0);
  auto const& h_feature_types = this->feature_types_.ConstHostSpan();
  for (bst_feature_t i = 0; i < num_columns_; ++i) {
    size_t column_size = std::max(static_cast<size_t>(1ul),
                                  this->Column(i).size());
    if (IsCat(h_feature_types, i)) {
      h_out_columns_ptr.push_back(static_cast<size_t>(column_size));
    } else {
      h_out_columns_ptr.push_back(std::min(static_cast<size_t>(column_size),
                                           static_cast<size_t>(num_bins_)));
    }
  }
  std::partial_sum(h_out_columns_ptr.begin(), h_out_columns_ptr.end(),
                   h_out_columns_ptr.begin());
  auto d_out_columns_ptr = p_cuts->cut_ptrs_.ConstDeviceSpan();

  // Set up output cuts
  size_t total_bins = h_out_columns_ptr.back();
  p_cuts->cut_values_.SetDevice(device_);
  p_cuts->cut_values_.Resize(total_bins);
  auto out_cut_values = p_cuts->cut_values_.DeviceSpan();
  auto d_ft = feature_types_.ConstDeviceSpan();

  dh::LaunchN(0, total_bins, [=] __device__(size_t idx) {
    auto column_id = dh::SegmentId(d_out_columns_ptr, idx);
    auto in_column = in_cut_values.subspan(d_in_columns_ptr[column_id],
                                           d_in_columns_ptr[column_id + 1] -
                                               d_in_columns_ptr[column_id]);
    auto out_column = out_cut_values.subspan(d_out_columns_ptr[column_id],
                                             d_out_columns_ptr[column_id + 1] -
                                                 d_out_columns_ptr[column_id]);
    idx -= d_out_columns_ptr[column_id];
    if (in_column.size() == 0) {
      // If the column is empty, we push a dummy value.  It won't affect training as the
      // column is empty, trees cannot split on it.  This is just to be consistent with
      // rest of the library.
      if (idx == 0) {
        d_min_values[column_id] = kRtEps;
        out_column[0] = kRtEps;
        assert(out_column.size() == 1);
      }
      return;
    }

    if (idx == 0 && !IsCat(d_ft, column_id)) {
      auto mval = in_column[idx].value;
      d_min_values[column_id] = mval - (fabs(mval) + 1e-5);
    }

    if (IsCat(d_ft, column_id)) {
      assert(out_column.size() == in_column.size());
      out_column[idx] = in_column[idx].value;
      return;
    }

    // Last thread is responsible for setting a value that's greater than other cuts.
    if (idx == out_column.size() - 1) {
      const bst_float cpt = in_column.back().value;
      // this must be bigger than last value in a scale
      const bst_float last = cpt + (fabs(cpt) + 1e-5);
      out_column[idx] = last;
      return;
    }
    assert(idx+1 < in_column.size());
    out_column[idx] = in_column[idx+1].value;
  });
  timer_.Stop(__func__);
}
}  // namespace common
}  // namespace xgboost
