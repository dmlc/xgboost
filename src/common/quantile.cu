#include "hip/hip_runtime.h"
/*!
 * Copyright 2020 by XGBoost Contributors
 */
#include <thrust/unique.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/binary_search.h>
#include <thrust/transform_scan.h>
#include <thrust/execution_policy.h>

#include <memory>
#include <utility>

#include "xgboost/span.h"
#include "quantile.h"
#include "quantile.cuh"
#include "hist_util.h"
#include "device_helpers.cuh"
#include "common.h"

namespace xgboost {
namespace common {

using WQSketch = HostSketchContainer::WQSketch;
using SketchEntry = WQSketch::Entry;

// Algorithm 4 in XGBoost's paper, using binary search to find i.
__device__ SketchEntry BinarySearchQuery(Span<SketchEntry const> const& entries, float rank) {
  assert(entries.size() >= 2);
  rank *= 2;
  if (rank < entries.front().rmin + entries.front().rmax) {
    return entries.front();
  }
  if (rank >= entries.back().rmin + entries.back().rmax) {
    return entries.back();
  }

  auto begin = dh::MakeTransformIterator<float>(
      entries.begin(), [=] __device__(SketchEntry const &entry) {
        return entry.rmin + entry.rmax;
      });
  auto end = begin + entries.size();
  auto i = thrust::upper_bound(thrust::seq, begin + 1, end - 1, rank) - begin - 1;
  if (rank < entries[i].RMinNext() + entries[i+1].RMaxPrev()) {
    return entries[i];
  } else {
    return entries[i+1];
  }
}

template <typename T>
void CopyTo(Span<T> out, Span<T const> src) {
  CHECK_EQ(out.size(), src.size());
  dh::safe_cuda(hipMemcpyAsync(out.data(), src.data(),
                                out.size_bytes(),
                                hipMemcpyDefault));
}

// Compute the merge path.
common::Span<thrust::tuple<uint64_t, uint64_t>> MergePath(
    Span<SketchEntry const> const &d_x, Span<bst_row_t const> const &x_ptr,
    Span<SketchEntry const> const &d_y, Span<bst_row_t const> const &y_ptr,
    Span<SketchEntry> out, Span<bst_row_t> out_ptr) {
  auto x_merge_key_it = thrust::make_zip_iterator(thrust::make_tuple(
      dh::MakeTransformIterator<bst_row_t>(
          thrust::make_counting_iterator(0ul),
          [=] __device__(size_t idx) { return dh::SegmentId(x_ptr, idx); }),
      d_x.data()));
  auto y_merge_key_it = thrust::make_zip_iterator(thrust::make_tuple(
      dh::MakeTransformIterator<bst_row_t>(
          thrust::make_counting_iterator(0ul),
          [=] __device__(size_t idx) { return dh::SegmentId(y_ptr, idx); }),
      d_y.data()));

  using Tuple = thrust::tuple<uint64_t, uint64_t>;

  thrust::constant_iterator<uint64_t> a_ind_iter(0ul);
  thrust::constant_iterator<uint64_t> b_ind_iter(1ul);

  auto place_holder = thrust::make_constant_iterator<uint64_t>(0u);
  auto x_merge_val_it =
      thrust::make_zip_iterator(thrust::make_tuple(a_ind_iter, place_holder));
  auto y_merge_val_it =
      thrust::make_zip_iterator(thrust::make_tuple(b_ind_iter, place_holder));

  dh::XGBCachingDeviceAllocator<Tuple> alloc;
  static_assert(sizeof(Tuple) == sizeof(SketchEntry), "");
  // We reuse the memory for storing merge path.
  common::Span<Tuple> merge_path{reinterpret_cast<Tuple *>(out.data()), out.size()};
  // Determine the merge path, 0 if element is from x, 1 if it's from y.
  thrust::merge_by_key(
      thrust::cuda::par(alloc), x_merge_key_it, x_merge_key_it + d_x.size(),
      y_merge_key_it, y_merge_key_it + d_y.size(), x_merge_val_it,
      y_merge_val_it, thrust::make_discard_iterator(), merge_path.data(),
      [=] __device__(auto const &l, auto const &r) -> bool {
        auto l_column_id = thrust::get<0>(l);
        auto r_column_id = thrust::get<0>(r);
        if (l_column_id == r_column_id) {
          return thrust::get<1>(l).value < thrust::get<1>(r).value;
        }
        return l_column_id < r_column_id;
      });

  // Compute output ptr
  auto transform_it =
      thrust::make_zip_iterator(thrust::make_tuple(x_ptr.data(), y_ptr.data()));
  thrust::transform(
      thrust::cuda::par(alloc), transform_it, transform_it + x_ptr.size(),
      out_ptr.data(),
      [] __device__(auto const& t) { return thrust::get<0>(t) + thrust::get<1>(t); });

  // 0^th is the indicator, 1^th is placeholder
  auto get_ind = []XGBOOST_DEVICE(Tuple const& t) { return thrust::get<0>(t); };
  // 0^th is the counter for x, 1^th for y.
  auto get_x =   []XGBOOST_DEVICE(Tuple const &t) { return thrust::get<0>(t); };
  auto get_y =   []XGBOOST_DEVICE(Tuple const &t) { return thrust::get<1>(t); };

  auto scan_key_it = dh::MakeTransformIterator<size_t>(
      thrust::make_counting_iterator(0ul),
      [=] __device__(size_t idx) { return dh::SegmentId(out_ptr, idx); });

  auto scan_val_it = dh::MakeTransformIterator<Tuple>(
      merge_path.data(), [=] __device__(Tuple const &t) -> Tuple {
        auto ind = get_ind(t);  // == 0 if element is from x
        // x_counter, y_counter
        return thrust::make_tuple<uint64_t, uint64_t>(!ind, ind);
      });

  // Compute the index for both x and y (which of the element in a and b are used in each
  // comparison) by scaning the binary merge path.  Take output [(x_0, y_0), (x_0, y_1),
  // ...] as an example, the comparison between (x_0, y_0) adds 1 step in the merge path.
  // Asumming y_0 is less than x_0 so this step is torward the end of y.  After the
  // comparison, index of y is incremented by 1 from y_0 to y_1, and at the same time, y_0
  // is landed into output as the first element in merge result.  The scan result is the
  // subscript of x and y.
  thrust::exclusive_scan_by_key(
      thrust::cuda::par(alloc), scan_key_it, scan_key_it + merge_path.size(),
      scan_val_it, merge_path.data(),
      thrust::make_tuple<uint64_t, uint64_t>(0ul, 0ul),
      thrust::equal_to<size_t>{},
      [=] __device__(Tuple const &l, Tuple const &r) -> Tuple {
        return thrust::make_tuple(get_x(l) + get_x(r), get_y(l) + get_y(r));
      });

  return merge_path;
}

// Merge d_x and d_y into out.  Because the final output depends on predicate (which
// summary does the output element come from) result by definition of merged rank.  So we
// run it in 2 passes to obtain the merge path and then customize the standard merge
// algorithm.
void MergeImpl(int32_t device, Span<SketchEntry const> const &d_x,
               Span<bst_row_t const> const &x_ptr,
               Span<SketchEntry const> const &d_y,
               Span<bst_row_t const> const &y_ptr,
               Span<SketchEntry> out,
               Span<bst_row_t> out_ptr) {
  dh::safe_cuda(hipSetDevice(device));
  CHECK_EQ(d_x.size() + d_y.size(), out.size());
  CHECK_EQ(x_ptr.size(), out_ptr.size());
  CHECK_EQ(y_ptr.size(), out_ptr.size());

  auto d_merge_path = MergePath(d_x, x_ptr, d_y, y_ptr, out, out_ptr);
  auto d_out = out;

  dh::LaunchN(device, d_out.size(), [=] __device__(size_t idx) {
    auto column_id = dh::SegmentId(out_ptr, idx);
    idx -= out_ptr[column_id];

    auto d_x_column =
        d_x.subspan(x_ptr[column_id], x_ptr[column_id + 1] - x_ptr[column_id]);
    auto d_y_column =
        d_y.subspan(y_ptr[column_id], y_ptr[column_id + 1] - y_ptr[column_id]);
    auto d_out_column = d_out.subspan(
        out_ptr[column_id], out_ptr[column_id + 1] - out_ptr[column_id]);
    auto d_path_column = d_merge_path.subspan(
        out_ptr[column_id], out_ptr[column_id + 1] - out_ptr[column_id]);

    uint64_t a_ind, b_ind;
    thrust::tie(a_ind, b_ind) = d_path_column[idx];

    // Handle empty column.  If both columns are empty, we should not get this column_id
    // as result of binary search.
    assert((d_x_column.size() != 0) || (d_y_column.size() != 0));
    if (d_x_column.size() == 0) {
      d_out_column[idx] = d_y_column[b_ind];
      return;
    }
    if (d_y_column.size() == 0) {
      d_out_column[idx] = d_x_column[a_ind];
      return;
    }

    // Handle trailing elements.
    assert(a_ind <= d_x_column.size());
    if (a_ind == d_x_column.size()) {
      // Trailing elements are from y because there's no more x to land.
      auto y_elem = d_y_column[b_ind];
      d_out_column[idx] = SketchEntry(y_elem.rmin + d_x_column.back().RMinNext(),
                                      y_elem.rmax + d_x_column.back().rmax,
                                      y_elem.wmin, y_elem.value);
      return;
    }
    auto x_elem = d_x_column[a_ind];
    assert(b_ind <= d_y_column.size());
    if (b_ind == d_y_column.size()) {
      d_out_column[idx] = SketchEntry(x_elem.rmin + d_y_column.back().RMinNext(),
                                      x_elem.rmax + d_y_column.back().rmax,
                                      x_elem.wmin, x_elem.value);
      return;
    }
    auto y_elem = d_y_column[b_ind];

    /* Merge procedure.  See A.3 merge operation eq (26) ~ (28).  The trick to interpret
       it is rewriting the symbols on both side of equality.  Take eq (26) as an example:
       Expand it according to definition of extended rank then rewrite it into:

       If $k_i$ is the $i$ element in output and \textbf{comes from $D_1$}:

         r_\bar{D}(k_i) = r_{\bar{D_1}}(k_i) + w_{\bar{{D_1}}}(k_i) +
                                          [r_{\bar{D_2}}(x_i) + w_{\bar{D_2}}(x_i)]

       Where $x_i$ is the largest element in $D_2$ that's less than $k_i$.  $k_i$ can be
       used in $D_1$ as it's since $k_i \in D_1$.  Other 2 equations can be applied
       similarly with $k_i$ comes from different $D$.  just use different symbol on
       different source of summary.
    */
    assert(idx < d_out_column.size());
    if (x_elem.value == y_elem.value) {
      d_out_column[idx] =
          SketchEntry{x_elem.rmin + y_elem.rmin, x_elem.rmax + y_elem.rmax,
                      x_elem.wmin + y_elem.wmin, x_elem.value};
    } else if (x_elem.value < y_elem.value) {
      // elem from x is landed. yprev_min is the element in D_2 that's 1 rank less than
      // x_elem if we put x_elem in D_2.
      float yprev_min = b_ind == 0 ? 0.0f : d_y_column[b_ind - 1].RMinNext();
      // rmin should be equal to x_elem.rmin + x_elem.wmin + yprev_min.  But for
      // implementation, the weight is stored in a separated field and we compute the
      // extended definition on the fly when needed.
      d_out_column[idx] =
          SketchEntry{x_elem.rmin + yprev_min, x_elem.rmax + y_elem.RMaxPrev(),
                      x_elem.wmin, x_elem.value};
    } else {
      // elem from y is landed.
      float xprev_min = a_ind == 0 ? 0.0f : d_x_column[a_ind - 1].RMinNext();
      d_out_column[idx] =
          SketchEntry{xprev_min + y_elem.rmin, x_elem.RMaxPrev() + y_elem.rmax,
                      y_elem.wmin, y_elem.value};
    }
  });
}

void SketchContainer::Push(common::Span<OffsetT const> cuts_ptr,
                           dh::caching_device_vector<SketchEntry>* entries) {
  timer_.Start(__func__);
  dh::safe_cuda(hipSetDevice(device_));
  // Copy or merge the new cuts, pruning is performed during `MakeCuts`.
  if (this->Current().size() == 0) {
    CHECK_EQ(this->columns_ptr_.Size(), cuts_ptr.size());
    // See thrust issue 1030, THRUST_CPP_DIALECT is not correctly defined so
    // move constructor is not used.
    this->Current().swap(*entries);
    CHECK_EQ(entries->size(), 0);
    auto d_cuts_ptr = this->columns_ptr_.DevicePointer();
    thrust::copy(thrust::device, cuts_ptr.data(),
                 cuts_ptr.data() + cuts_ptr.size(), d_cuts_ptr);
  } else {
    auto d_entries = dh::ToSpan(*entries);
    this->Merge(cuts_ptr, d_entries);
    this->FixError();
  }
  CHECK_NE(this->columns_ptr_.Size(), 0);
  timer_.Stop(__func__);
}

size_t SketchContainer::Unique() {
  timer_.Start(__func__);
  dh::safe_cuda(hipSetDevice(device_));
  this->columns_ptr_.SetDevice(device_);
  Span<OffsetT> d_column_scan = this->columns_ptr_.DeviceSpan();
  CHECK_EQ(d_column_scan.size(), num_columns_ + 1);
  Span<SketchEntry> entries = dh::ToSpan(this->Current());
  HostDeviceVector<OffsetT> scan_out(d_column_scan.size());
  scan_out.SetDevice(device_);
  auto d_scan_out = scan_out.DeviceSpan();

  d_column_scan = this->columns_ptr_.DeviceSpan();
  size_t n_uniques = dh::SegmentedUnique(
      d_column_scan.data(), d_column_scan.data() + d_column_scan.size(),
      entries.data(), entries.data() + entries.size(), scan_out.DevicePointer(),
      entries.data(),
      detail::SketchUnique{});
  this->columns_ptr_.Copy(scan_out);
  CHECK(!this->columns_ptr_.HostCanRead());

  this->Current().resize(n_uniques);
  timer_.Stop(__func__);
  return n_uniques;
}

void SketchContainer::Prune(size_t to) {
  timer_.Start(__func__);
  dh::safe_cuda(hipSetDevice(device_));

  this->Unique();
  OffsetT to_total = 0;
  HostDeviceVector<OffsetT> new_columns_ptr{to_total};
  for (bst_feature_t i = 0; i < num_columns_; ++i) {
    size_t length = this->Column(i).size();
    length = std::min(length, to);
    to_total += length;
    new_columns_ptr.HostVector().emplace_back(to_total);
  }
  new_columns_ptr.SetDevice(device_);
  this->Other().resize(to_total);

  auto d_columns_ptr_in = this->columns_ptr_.ConstDeviceSpan();
  auto d_columns_ptr_out = new_columns_ptr.ConstDeviceSpan();
  auto out = dh::ToSpan(this->Other());
  auto in = dh::ToSpan(this->Current());
  dh::LaunchN(0, to_total, [=] __device__(size_t idx) {
    size_t column_id = dh::SegmentId(d_columns_ptr_out, idx);
    auto out_column = out.subspan(d_columns_ptr_out[column_id],
                                  d_columns_ptr_out[column_id + 1] -
                                      d_columns_ptr_out[column_id]);
    auto in_column = in.subspan(d_columns_ptr_in[column_id],
                                d_columns_ptr_in[column_id + 1] -
                                    d_columns_ptr_in[column_id]);
    idx -= d_columns_ptr_out[column_id];
    // Input has lesser columns than `to`, just copy them to the output.  This is correct
    // as the new output size is calculated based on both the size of `to` and current
    // column.
    if (in_column.size() <= to) {
      out_column[idx] = in_column[idx];
      return;
    }
    // 1 thread for each output.  See A.4 for detail.
    auto entries = in_column;
    auto d_out = out_column;
    if (idx == 0) {
      d_out.front() = entries.front();
      return;
    }
    if (idx == to - 1) {
      d_out.back() = entries.back();
      return;
    }

    float w = entries.back().rmin - entries.front().rmax;
    assert(w != 0);
    auto budget = static_cast<float>(d_out.size());
    assert(budget != 0);
    auto q = ((idx * w) / (to - 1) + entries.front().rmax);
    d_out[idx] = BinarySearchQuery(entries, q);
  });
  this->columns_ptr_.HostVector() = new_columns_ptr.HostVector();
  this->Alternate();
  timer_.Stop(__func__);
}

void SketchContainer::Merge(Span<OffsetT const> d_that_columns_ptr,
                            Span<SketchEntry const> that) {
  dh::safe_cuda(hipSetDevice(device_));
  timer_.Start(__func__);
  if (this->Current().size() == 0) {
    CHECK_EQ(this->columns_ptr_.HostVector().back(), 0);
    CHECK_EQ(this->columns_ptr_.HostVector().size(), d_that_columns_ptr.size());
    CHECK_EQ(columns_ptr_.Size(), num_columns_ + 1);
    thrust::copy(thrust::device, d_that_columns_ptr.data(),
                 d_that_columns_ptr.data() + d_that_columns_ptr.size(),
                 this->columns_ptr_.DevicePointer());
    auto total = this->columns_ptr_.HostVector().back();
    this->Current().resize(total);
    CopyTo(dh::ToSpan(this->Current()), that);
    timer_.Stop(__func__);
    return;
  }

  this->Other().resize(this->Current().size() + that.size());
  CHECK_EQ(d_that_columns_ptr.size(), this->columns_ptr_.Size());

  HostDeviceVector<OffsetT> new_columns_ptr;
  new_columns_ptr.SetDevice(device_);
  new_columns_ptr.Resize(this->ColumnsPtr().size());
  MergeImpl(device_, this->Data(), this->ColumnsPtr(),
            that, d_that_columns_ptr,
            dh::ToSpan(this->Other()), new_columns_ptr.DeviceSpan());
  this->columns_ptr_ = std::move(new_columns_ptr);
  CHECK_EQ(this->columns_ptr_.Size(), num_columns_ + 1);
  CHECK_EQ(new_columns_ptr.Size(), 0);
  this->Alternate();
  timer_.Stop(__func__);
}

void SketchContainer::FixError() {
  dh::safe_cuda(hipSetDevice(device_));
  auto d_columns_ptr = this->columns_ptr_.ConstDeviceSpan();
  auto in = dh::ToSpan(this->Current());
  dh::LaunchN(device_, in.size(), [=] __device__(size_t idx) {
    auto column_id = dh::SegmentId(d_columns_ptr, idx);
    auto in_column = in.subspan(d_columns_ptr[column_id],
                                d_columns_ptr[column_id + 1] -
                                    d_columns_ptr[column_id]);
    idx -= d_columns_ptr[column_id];
    float prev_rmin = idx == 0 ? 0.0f : in_column[idx-1].rmin;
    if (in_column[idx].rmin < prev_rmin) {
      in_column[idx].rmin = prev_rmin;
    }
    float prev_rmax = idx == 0 ? 0.0f : in_column[idx-1].rmax;
    if (in_column[idx].rmax < prev_rmax) {
      in_column[idx].rmax = prev_rmax;
    }
    float rmin_next = in_column[idx].RMinNext();
    if (in_column[idx].rmax < rmin_next) {
      in_column[idx].rmax = rmin_next;
    }
  });
}

void SketchContainer::AllReduce() {
  dh::safe_cuda(hipSetDevice(device_));
  auto world = rabit::GetWorldSize();
  if (world == 1) {
    return;
  }

  timer_.Start(__func__);
  if (!reducer_) {
    reducer_ = std::make_unique<dh::AllReducer>();
    reducer_->Init(device_);
  }
  // Reduce the overhead on syncing.
  size_t global_sum_rows = num_rows_;
  rabit::Allreduce<rabit::op::Sum>(&global_sum_rows, 1);
  size_t intermediate_num_cuts =
      std::min(global_sum_rows, static_cast<size_t>(num_bins_ * kFactor));
  this->Prune(intermediate_num_cuts);

  auto d_columns_ptr = this->columns_ptr_.ConstDeviceSpan();
  CHECK_EQ(d_columns_ptr.size(), num_columns_ + 1);
  size_t n = d_columns_ptr.size();
  rabit::Allreduce<rabit::op::Max>(&n, 1);
  CHECK_EQ(n, d_columns_ptr.size()) << "Number of columns differs across workers";

  // Get the columns ptr from all workers
  dh::device_vector<SketchContainer::OffsetT> gathered_ptrs;
  gathered_ptrs.resize(d_columns_ptr.size() * world, 0);
  size_t rank = rabit::GetRank();
  auto offset = rank * d_columns_ptr.size();
  thrust::copy(thrust::device, d_columns_ptr.data(), d_columns_ptr.data() + d_columns_ptr.size(),
               gathered_ptrs.begin() + offset);
  reducer_->AllReduceSum(gathered_ptrs.data().get(), gathered_ptrs.data().get(),
                         gathered_ptrs.size());

  // Get the data from all workers.
  std::vector<size_t> recv_lengths;
  dh::caching_device_vector<char> recvbuf;
  reducer_->AllGather(this->Current().data().get(),
                      dh::ToSpan(this->Current()).size_bytes(), &recv_lengths,
                      &recvbuf);
  reducer_->Synchronize();

  // Segment the received data.
  auto s_recvbuf = dh::ToSpan(recvbuf);
  std::vector<Span<SketchEntry>> allworkers;
  offset = 0;
  for (int32_t i = 0; i < world; ++i) {
    size_t length_as_bytes = recv_lengths.at(i);
    auto raw = s_recvbuf.subspan(offset, length_as_bytes);
    auto sketch = Span<SketchEntry>(reinterpret_cast<SketchEntry *>(raw.data()),
                                    length_as_bytes / sizeof(SketchEntry));
    allworkers.emplace_back(sketch);
    offset += length_as_bytes;
  }

  // Merge them into a new sketch.
  SketchContainer new_sketch(num_bins_, this->num_columns_, global_sum_rows,
                             this->device_);
  for (size_t i = 0; i < allworkers.size(); ++i) {
    auto worker = allworkers[i];
    auto worker_ptr =
        dh::ToSpan(gathered_ptrs)
            .subspan(i * d_columns_ptr.size(), d_columns_ptr.size());
    new_sketch.Merge(worker_ptr, worker);
    new_sketch.FixError();
  }

  *this = std::move(new_sketch);
  timer_.Stop(__func__);
}

void SketchContainer::MakeCuts(HistogramCuts* p_cuts) {
  timer_.Start(__func__);
  dh::safe_cuda(hipSetDevice(device_));
  p_cuts->min_vals_.Resize(num_columns_);

  // Sync between workers.
  this->AllReduce();

  // Prune to final number of bins.
  this->Prune(num_bins_ + 1);
  this->Unique();
  this->FixError();

  // Set up inputs
  auto d_in_columns_ptr = this->columns_ptr_.ConstDeviceSpan();

  p_cuts->min_vals_.SetDevice(device_);
  auto d_min_values = p_cuts->min_vals_.DeviceSpan();
  auto in_cut_values = dh::ToSpan(this->Current());

  // Set up output ptr
  p_cuts->cut_ptrs_.SetDevice(device_);
  auto& h_out_columns_ptr = p_cuts->cut_ptrs_.HostVector();
  h_out_columns_ptr.clear();
  h_out_columns_ptr.push_back(0);
  for (bst_feature_t i = 0; i < num_columns_; ++i) {
    h_out_columns_ptr.push_back(
        std::min(static_cast<size_t>(std::max(static_cast<size_t>(1ul),
                                              this->Column(i).size())),
                 static_cast<size_t>(num_bins_)));
  }
  std::partial_sum(h_out_columns_ptr.begin(), h_out_columns_ptr.end(),
                   h_out_columns_ptr.begin());
  auto d_out_columns_ptr = p_cuts->cut_ptrs_.ConstDeviceSpan();

  // Set up output cuts
  size_t total_bins = h_out_columns_ptr.back();
  p_cuts->cut_values_.SetDevice(device_);
  p_cuts->cut_values_.Resize(total_bins);
  auto out_cut_values = p_cuts->cut_values_.DeviceSpan();

  dh::LaunchN(0, total_bins, [=] __device__(size_t idx) {
    auto column_id = dh::SegmentId(d_out_columns_ptr, idx);
    auto in_column = in_cut_values.subspan(d_in_columns_ptr[column_id],
                                           d_in_columns_ptr[column_id + 1] -
                                               d_in_columns_ptr[column_id]);
    auto out_column = out_cut_values.subspan(d_out_columns_ptr[column_id],
                                             d_out_columns_ptr[column_id + 1] -
                                                 d_out_columns_ptr[column_id]);
    idx -= d_out_columns_ptr[column_id];
    if (in_column.size() == 0) {
      // If the column is empty, we push a dummy value.  It won't affect training as the
      // column is empty, trees cannot split on it.  This is just to be consistent with
      // rest of the library.
      if (idx == 0) {
        d_min_values[column_id] = kRtEps;
        out_column[0] = kRtEps;
        assert(out_column.size() == 1);
      }
      return;
    }

    // First thread is responsible for setting min values.
    if (idx == 0) {
      auto mval = in_column[idx].value;
      d_min_values[column_id] = mval - (fabs(mval) + 1e-5);
    }
    // Last thread is responsible for setting a value that's greater than other cuts.
    if (idx == out_column.size() - 1) {
      const bst_float cpt = in_column.back().value;
      // this must be bigger than last value in a scale
      const bst_float last = cpt + (fabs(cpt) + 1e-5);
      out_column[idx] = last;
      return;
    }
    assert(idx+1 < in_column.size());
    out_column[idx] = in_column[idx+1].value;
  });
  timer_.Stop(__func__);
}
}  // namespace common
}  // namespace xgboost
