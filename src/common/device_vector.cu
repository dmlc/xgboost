/**
 * Copyright 2017-2024, XGBoost contributors
 */
#include "../collective/communicator-inl.h"  // for GetRank
#include "common.h"                          // for HumanMemUnit
#include "cuda_dr_utils.h"
#include "device_helpers.cuh"  // for CurrentDevice
#include "device_vector.cuh"

namespace dh {
namespace detail {
void ThrowOOMError(std::string const &err, std::size_t bytes) {
  auto device = CurrentDevice();
  auto rank = xgboost::collective::GetRank();
  using xgboost::common::HumanMemUnit;
  std::stringstream ss;
  ss << "Memory allocation error on worker " << rank << ": " << err << "\n"
     << "- Free memory: " << HumanMemUnit(dh::AvailableMemory(device)) << "\n"
     << "- Requested memory: " << HumanMemUnit(bytes) << std::endl;
  LOG(FATAL) << ss.str();
}

GrowOnlyVirtualMemVec::GrowOnlyVirtualMemVec(hipMemLocationType type)
    : prop_{xgboost::cudr::MakeAllocProp(type)} {
  CHECK(type == hipMemLocationTypeDevice || type == hipMemLocationTypeHostNuma);
  // Get the allocation granularity.
  this->granularity_ = xgboost::cudr::GetAllocGranularity(&this->prop_);
  auto ordinal = CurrentDevice();

  // Assign the access descriptor
  hipMemAccessDesc dacc;
  dacc.flags = hipMemAccessFlagsProtReadWrite;
  dacc.location.type = hipMemLocationTypeDevice;
  dacc.location.id = ordinal;
  this->access_desc_.push_back(dacc);

  if (type == hipMemLocationTypeHostNuma) {
    hipMemAccessDesc hacc;
    hacc.flags = hipMemAccessFlagsProtReadWrite;

    xgboost::cudr::GetCuLocation(hipMemLocationTypeHostNuma, &hacc.location);
    this->access_desc_.push_back(hacc);
  }
}
}  // namespace detail

#if defined(XGBOOST_USE_RMM)
LoggingResource *GlobalLoggingResource() {
  static auto mr{std::make_unique<LoggingResource>()};
  return mr.get();
}
#endif  // defined(XGBOOST_USE_RMM)
}  // namespace dh
