/**
 * Copyright 2025, XGBoost Contributors
 */
#include "cuda_pinned_allocator.h"

#if defined(XGBOOST_USE_CUDA)

#include <hip/hip_runtime_api.h>  // for hipMemPoolCreate, hipMemPoolDestroy

#include <array>    // for array
#include <cstring>  // for memset
#include <memory>   // for unique_ptr

#endif  // defined(XGBOOST_USE_CUDA)

#include "common.h"
#include "cuda_dr_utils.h"  // for CUDA_HW_DECOM_AVAILABLE
#include "cuda_rt_utils.h"  // for CurrentDevice

namespace xgboost::common::cuda_impl {
[[nodiscard]] MemPoolHdl CreateHostMemPool() {
  auto mem_pool = std::unique_ptr<hipMemPool_t, void (*)(hipMemPool_t*)>{
      [] {
        hipMemPoolProps h_props;
        std::memset(&h_props, '\0', sizeof(h_props));
        auto numa_id = curt::GetNumaId();
        h_props.location.id = numa_id;
        h_props.location.type = hipMemLocationTypeHostNuma;
        h_props.allocType = hipMemAllocationTypePinned;
#if defined(CUDA_HW_DECOM_AVAILABLE)
        h_props.usage = cudaMemPoolCreateUsageHwDecompress;
#endif  // defined(CUDA_HW_DECOM_AVAILABLE)
        h_props.handleTypes = hipMemHandleTypeNone;

        hipMemPoolProps d_props;
        std::memset(&d_props, '\0', sizeof(d_props));
        auto device_idx = curt::CurrentDevice();
        d_props.location.id = device_idx;
        d_props.location.type = hipMemLocationTypeDevice;
        d_props.allocType = hipMemAllocationTypePinned;
#if defined(CUDA_HW_DECOM_AVAILABLE)
        d_props.usage = cudaMemPoolCreateUsageHwDecompress;
#endif  // defined(CUDA_HW_DECOM_AVAILABLE)
        d_props.handleTypes = hipMemHandleTypeNone;

        std::array<hipMemPoolProps, 2> vprops{h_props, d_props};

        hipMemPool_t* mem_pool = new hipMemPool_t;
        dh::safe_cuda(hipMemPoolCreate(mem_pool, vprops.data()));

        hipMemAccessDesc h_desc;
        h_desc.location = h_props.location;
        h_desc.flags = hipMemAccessFlagsProtReadWrite;

        hipMemAccessDesc d_desc;
        d_desc.location = d_props.location;
        d_desc.flags = hipMemAccessFlagsProtReadWrite;

        std::array<hipMemAccessDesc, 2> descs{h_desc, d_desc};
        dh::safe_cuda(hipMemPoolSetAccess(*mem_pool, descs.data(), descs.size()));
        return mem_pool;
      }(),
      [](hipMemPool_t* mem_pool) {
        if (mem_pool) {
          dh::safe_cuda(hipMemPoolDestroy(*mem_pool));
        }
      }};
  return mem_pool;
}
}  // namespace xgboost::common::cuda_impl
