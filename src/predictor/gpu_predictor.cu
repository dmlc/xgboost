#include "hip/hip_runtime.h"
/*!
 * Copyright by Contributors 2017
 */
#include <dmlc/parameter.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <xgboost/data.h>
#include <xgboost/predictor.h>
#include <xgboost/tree_model.h>
#include <xgboost/tree_updater.h>
#include <memory>
#include "../common/device_helpers.cuh"

namespace xgboost {
namespace predictor {

DMLC_REGISTRY_FILE_TAG(gpu_predictor);

/*! \brief prediction parameters */
struct GPUPredictionParam : public dmlc::Parameter<GPUPredictionParam> {
  int gpu_id;
  int n_gpus;
  bool silent;
  // declare parameters
  DMLC_DECLARE_PARAMETER(GPUPredictionParam) {
    DMLC_DECLARE_FIELD(gpu_id).set_default(0).describe(
        "Device ordinal for GPU prediction.");
    DMLC_DECLARE_FIELD(n_gpus).set_default(1).describe(
        "Number of devices to use for prediction (NOT IMPLEMENTED).");
    DMLC_DECLARE_FIELD(silent).set_default(false).describe(
        "Do not print information during trainig.");
  }
};
DMLC_REGISTER_PARAMETER(GPUPredictionParam);

template <typename iter_t>
void increment_offset(iter_t begin_itr, iter_t end_itr, size_t amount) {
  thrust::transform(begin_itr, end_itr, begin_itr,
                    [=] __device__(size_t elem) { return elem + amount; });
}

/**
 * \struct  DeviceMatrix
 *
 * \brief A csr representation of the input matrix allocated on the device.
 */

struct DeviceMatrix {
  DMatrix* p_mat;  // Pointer to the original matrix on the host
  dh::bulk_allocator<dh::memory_type::DEVICE> ba;
  dh::dvec<size_t> row_ptr;
  dh::dvec<SparseBatch::Entry> data;
  thrust::device_vector<float> predictions;

  DeviceMatrix(DMatrix* dmat, int device_idx, bool silent) : p_mat(dmat) {
    dh::safe_cuda(hipSetDevice(device_idx));
    auto info = dmat->info();
    ba.allocate(device_idx, silent, &row_ptr, info.num_row + 1, &data,
                info.num_nonzero);
    auto iter = dmat->RowIterator();
    iter->BeforeFirst();
    size_t data_offset = 0;
    while (iter->Next()) {
      auto batch = iter->Value();
      // Copy row ptr
      thrust::copy(batch.ind_ptr, batch.ind_ptr + batch.size + 1,
                   row_ptr.tbegin() + batch.base_rowid);
      if (batch.base_rowid > 0) {
        auto begin_itr = row_ptr.tbegin() + batch.base_rowid;
        auto end_itr = begin_itr + batch.size + 1;
        increment_offset(begin_itr, end_itr, batch.base_rowid);
      }
      // Copy data
      thrust::copy(batch.data_ptr, batch.data_ptr + batch.ind_ptr[batch.size],
                   data.tbegin() + data_offset);
      data_offset += batch.ind_ptr[batch.size];
    }
  }
};

/**
 * \struct  DevicePredictionNode
 *
 * \brief Packed 16 byte representation of a tree node for use in device
 * prediction
 */

struct DevicePredictionNode {
  XGBOOST_DEVICE DevicePredictionNode()
      : fidx(-1), left_child_idx(-1), right_child_idx(-1) {}

  union NodeValue {
    float leaf_weight;
    float fvalue;
  };

  int fidx;
  int left_child_idx;
  int right_child_idx;
  NodeValue val;

  DevicePredictionNode(const RegTree::Node& n) {  // NOLINT
    this->left_child_idx = n.cleft();
    this->right_child_idx = n.cright();
    this->fidx = n.split_index();
    if (n.default_left()) {
      fidx |= (1U << 31);
    }

    if (n.is_leaf()) {
      this->val.leaf_weight = n.leaf_value();
    } else {
      this->val.fvalue = n.split_cond();
    }
  }

  XGBOOST_DEVICE bool IsLeaf() const { return left_child_idx == -1; }

  XGBOOST_DEVICE int GetFidx() const { return fidx & ((1U << 31) - 1U); }

  XGBOOST_DEVICE bool MissingLeft() const { return (fidx >> 31) != 0; }

  XGBOOST_DEVICE int MissingIdx() const {
    if (MissingLeft()) {
      return this->left_child_idx;
    } else {
      return this->right_child_idx;
    }
  }

  XGBOOST_DEVICE float GetFvalue() const { return val.fvalue; }

  XGBOOST_DEVICE float GetWeight() const { return val.leaf_weight; }
};

struct ElementLoader {
  bool use_shared;
  size_t* d_row_ptr;
  SparseBatch::Entry* d_data;
  int num_features;
  float* smem;

  __device__ ElementLoader(bool use_shared, size_t* row_ptr,
                           SparseBatch::Entry* entry, int num_features,
                           float* smem, int num_rows)
      : use_shared(use_shared),
        d_row_ptr(row_ptr),
        d_data(entry),
        num_features(num_features),
        smem(smem) {
    // Copy instances
    if (use_shared) {
      bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
      int shared_elements = blockDim.x * num_features;
      dh::block_fill(smem, shared_elements, nanf(""));
      __syncthreads();
      if (global_idx < num_rows) {
        bst_uint elem_begin = d_row_ptr[global_idx];
        bst_uint elem_end = d_row_ptr[global_idx + 1];
        for (bst_uint elem_idx = elem_begin; elem_idx < elem_end; elem_idx++) {
          SparseBatch::Entry elem = d_data[elem_idx];
          smem[threadIdx.x * num_features + elem.index] = elem.fvalue;
        }
      }
      __syncthreads();
    }
  }
  __device__ float GetFvalue(int ridx, int fidx) {
    if (use_shared) {
      return smem[threadIdx.x * num_features + fidx];
    } else {
      // Binary search
      auto begin_ptr = d_data + d_row_ptr[ridx];
      auto end_ptr = d_data + d_row_ptr[ridx + 1];
      SparseBatch::Entry* previous_middle = nullptr;
      while (end_ptr != begin_ptr) {
        auto middle = begin_ptr + (end_ptr - begin_ptr) / 2;
        if (middle == previous_middle) {
          break;
        } else {
          previous_middle = middle;
        }

        if (middle->index == fidx) {
          return middle->fvalue;
        } else if (middle->index < fidx) {
          begin_ptr = middle;
        } else {
          end_ptr = middle;
        }
      }
      // Value is missing
      return nanf("");
    }
  }
};

__device__ float GetLeafWeight(bst_uint ridx, const DevicePredictionNode* tree,
                               ElementLoader* loader) {
  DevicePredictionNode n = tree[0];
  while (!n.IsLeaf()) {
    float fvalue = loader->GetFvalue(ridx, n.GetFidx());
    // Missing value
    if (isnan(fvalue)) {
      n = tree[n.MissingIdx()];
    } else {
      if (fvalue < n.GetFvalue()) {
        n = tree[n.left_child_idx];
      } else {
        n = tree[n.right_child_idx];
      }
    }
  }
  return n.GetWeight();
}

template <int BLOCK_THREADS>
__global__ void PredictKernel(const DevicePredictionNode* d_nodes,
                              float* d_out_predictions, int* d_tree_segments,
                              int* d_tree_group, size_t* d_row_ptr,
                              SparseBatch::Entry* d_data, int tree_begin,
                              int tree_end, int num_features, bst_uint num_rows,
                              bool use_shared, int num_group) {
  extern __shared__ float smem[];
  bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
  ElementLoader loader(use_shared, d_row_ptr, d_data, num_features, smem,
                       num_rows);
  if (global_idx >= num_rows) return;
  if (num_group == 1) {
    float sum = 0;
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      const DevicePredictionNode* d_tree =
          d_nodes + d_tree_segments[tree_idx - tree_begin];
      sum += GetLeafWeight(global_idx, d_tree, &loader);
    }
    d_out_predictions[global_idx] += sum;
  } else {
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      int tree_group = d_tree_group[tree_idx];
      const DevicePredictionNode* d_tree =
          d_nodes + d_tree_segments[tree_idx - tree_begin];
      bst_uint out_prediction_idx = global_idx * num_group + tree_group;
      d_out_predictions[out_prediction_idx] +=
          GetLeafWeight(global_idx, d_tree, &loader);
    }
  }
}

class GPUPredictor : public xgboost::Predictor {
 private:
  void DevicePredictInternal(DMatrix* dmat, std::vector<bst_float>* out_preds,
                             const gbm::GBTreeModel& model, int tree_begin,
                             int tree_end) {
    if (tree_end - tree_begin == 0) {
      return;
    }

    // Add dmatrix to device if not seen before
    if (this->device_matrix_cache_.find(dmat) ==
        this->device_matrix_cache_.end()) {
      this->device_matrix_cache_.emplace(
          dmat, std::unique_ptr<DeviceMatrix>(
                    new DeviceMatrix(dmat, param.gpu_id, param.silent)));
    }
    DeviceMatrix* device_matrix = device_matrix_cache_.find(dmat)->second.get();

    dh::safe_cuda(hipSetDevice(param.gpu_id));
    CHECK_EQ(model.param.size_leaf_vector, 0);
    // Copy decision trees to device
    thrust::host_vector<int> h_tree_segments;
    h_tree_segments.reserve((tree_end - tree_end) + 1);
    int sum = 0;
    h_tree_segments.push_back(sum);
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      sum += model.trees[tree_idx]->GetNodes().size();
      h_tree_segments.push_back(sum);
    }

    thrust::host_vector<DevicePredictionNode> h_nodes(h_tree_segments.back());
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      auto& src_nodes = model.trees[tree_idx]->GetNodes();
      std::copy(src_nodes.begin(), src_nodes.end(),
                h_nodes.begin() + h_tree_segments[tree_idx - tree_begin]);
    }

    nodes.resize(h_nodes.size());
    thrust::copy(h_nodes.begin(), h_nodes.end(), nodes.begin());
    tree_segments.resize(h_tree_segments.size());
    thrust::copy(h_tree_segments.begin(), h_tree_segments.end(),
                 tree_segments.begin());
    tree_group.resize(model.tree_info.size());
    thrust::copy(model.tree_info.begin(), model.tree_info.end(),
                 tree_group.begin());

    if (device_matrix->predictions.size() != out_preds->size()) {
      device_matrix->predictions.resize(out_preds->size());
      thrust::copy(out_preds->begin(), out_preds->end(),
                   device_matrix->predictions.begin());
    }

    const int BLOCK_THREADS = 128;
    const int GRID_SIZE =
        dh::div_round_up(device_matrix->row_ptr.size() - 1, BLOCK_THREADS);

    int shared_memory_bytes =
        sizeof(float) * device_matrix->p_mat->info().num_col * BLOCK_THREADS;
    bool use_shared = true;
    if (shared_memory_bytes > dh::max_shared_memory(param.gpu_id)) {
      shared_memory_bytes = 0;
      use_shared = false;
    }

    PredictKernel<BLOCK_THREADS>
        <<<GRID_SIZE, BLOCK_THREADS, shared_memory_bytes>>>(
            dh::raw(nodes), dh::raw(device_matrix->predictions),
            dh::raw(tree_segments), dh::raw(tree_group),
            device_matrix->row_ptr.data(), device_matrix->data.data(),
            tree_begin, tree_end, device_matrix->p_mat->info().num_col,
            device_matrix->p_mat->info().num_row, use_shared,
            model.param.num_output_group);

    dh::safe_cuda(hipDeviceSynchronize());
    thrust::copy(device_matrix->predictions.begin(),
                 device_matrix->predictions.end(), out_preds->begin());
  }

 public:
  GPUPredictor() : cpu_predictor(Predictor::Create("cpu_predictor")) {}

  void PredictBatch(DMatrix* dmat, std::vector<bst_float>* out_preds,
                    const gbm::GBTreeModel& model, int tree_begin,
                    unsigned ntree_limit = 0) override {
    if (this->PredictFromCache(dmat, out_preds, model, ntree_limit)) {
      return;
    }
    this->InitOutPredictions(dmat->info(), out_preds, model);

    int tree_end = ntree_limit * model.param.num_output_group;
    if (ntree_limit == 0 || ntree_limit > model.trees.size()) {
      tree_end = static_cast<unsigned>(model.trees.size());
    }

    DevicePredictInternal(dmat, out_preds, model, tree_begin, tree_end);
  }

  void UpdatePredictionCache(
      const gbm::GBTreeModel& model,
      std::vector<std::unique_ptr<TreeUpdater>>* updaters,
      int num_new_trees) override {
    // dh::Timer t;
    int old_ntree = model.trees.size() - num_new_trees;
    // update cache entry
    for (auto& kv : cache_) {
      PredictionCacheEntry& e = kv.second;
      DMatrix* dmat = kv.first;

      if (e.predictions.size() == 0) {
        cpu_predictor->PredictBatch(dmat, &(e.predictions), model, 0,
                                    model.trees.size());
      } else if (model.param.num_output_group == 1 && updaters->size() > 0 &&
                 num_new_trees == 1 &&
                 updaters->back()->UpdatePredictionCache(e.data.get(),
                                                         &(e.predictions))) {
        {}  // do nothing
      } else {
        DevicePredictInternal(dmat, &(e.predictions), model, old_ntree,
                              model.trees.size());
      }
    }
  }

  void PredictInstance(const SparseBatch::Inst& inst,
                       std::vector<bst_float>* out_preds,
                       const gbm::GBTreeModel& model, unsigned ntree_limit,
                       unsigned root_index) override {
    cpu_predictor->PredictInstance(inst, out_preds, model, root_index);
  }
  void PredictLeaf(DMatrix* p_fmat, std::vector<bst_float>* out_preds,
                   const gbm::GBTreeModel& model,
                   unsigned ntree_limit) override {
    cpu_predictor->PredictLeaf(p_fmat, out_preds, model, ntree_limit);
  }

  void PredictContribution(DMatrix* p_fmat,
                           std::vector<bst_float>* out_contribs,
                           const gbm::GBTreeModel& model,
                           unsigned ntree_limit) override {
    cpu_predictor->PredictContribution(p_fmat, out_contribs, model,
                                       ntree_limit);
  }

  void Init(const std::vector<std::pair<std::string, std::string>>& cfg,
            const std::vector<std::shared_ptr<DMatrix>>& cache) override {
    Predictor::Init(cfg, cache);
    cpu_predictor->Init(cfg, cache);
    param.InitAllowUnknown(cfg);
  }

 private:
  GPUPredictionParam param;
  std::unique_ptr<Predictor> cpu_predictor;
  std::unordered_map<DMatrix*, std::unique_ptr<DeviceMatrix>>
      device_matrix_cache_;
  thrust::device_vector<DevicePredictionNode> nodes;
  thrust::device_vector<int> tree_segments;
  thrust::device_vector<int> tree_group;
};
XGBOOST_REGISTER_PREDICTOR(GPUPredictor, "gpu_predictor")
    .describe("Make predictions using GPU.")
    .set_body([]() { return new GPUPredictor(); });
}  // namespace predictor
}  // namespace xgboost
