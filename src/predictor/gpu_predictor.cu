#include "hip/hip_runtime.h"
/*!
 * Copyright by Contributors 2017
 */
#include <dmlc/parameter.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <xgboost/data.h>
#include <xgboost/predictor.h>
#include <xgboost/tree_model.h>
#include <xgboost/tree_updater.h>
#include <memory>
#include "../common/device_helpers.cuh"
#include "../common/host_device_vector.h"

namespace xgboost {
namespace predictor {

DMLC_REGISTRY_FILE_TAG(gpu_predictor);

/*! \brief prediction parameters */
struct GPUPredictionParam : public dmlc::Parameter<GPUPredictionParam> {
  int gpu_id;
  int n_gpus;
  bool silent;
  // declare parameters
  DMLC_DECLARE_PARAMETER(GPUPredictionParam) {
    DMLC_DECLARE_FIELD(gpu_id).set_default(0).describe(
        "Device ordinal for GPU prediction.");
    DMLC_DECLARE_FIELD(n_gpus).set_default(1).describe(
        "Number of devices to use for prediction (NOT IMPLEMENTED).");
    DMLC_DECLARE_FIELD(silent).set_default(false).describe(
        "Do not print information during trainig.");
  }
};
DMLC_REGISTER_PARAMETER(GPUPredictionParam);

template <typename IterT>
void IncrementOffset(IterT begin_itr, IterT end_itr, size_t amount) {
  thrust::transform(begin_itr, end_itr, begin_itr,
                    [=] __device__(size_t elem) { return elem + amount; });
}

/**
 * \struct  DeviceMatrix
 *
 * \brief A csr representation of the input matrix allocated on the device.
 */

struct DeviceMatrix {
  DMatrix* p_mat;  // Pointer to the original matrix on the host
  dh::BulkAllocator<dh::MemoryType::kDevice> ba;
  dh::DVec<size_t> row_ptr;
  dh::DVec<Entry> data;
  thrust::device_vector<float> predictions;

  DeviceMatrix(DMatrix* dmat, int device_idx, bool silent) : p_mat(dmat) {
    dh::safe_cuda(hipSetDevice(device_idx));
    auto info = dmat->Info();
    ba.Allocate(device_idx, silent, &row_ptr, info.num_row_ + 1, &data,
                info.num_nonzero_);
    auto iter = dmat->RowIterator();
    iter->BeforeFirst();
    size_t data_offset = 0;
    while (iter->Next()) {
      auto &batch = iter->Value();
      // Copy row ptr
      dh::safe_cuda(hipMemcpy(
          row_ptr.Data() + batch.base_rowid, batch.offset.data(),
          sizeof(size_t) * batch.offset.size(), hipMemcpyHostToDevice));
      if (batch.base_rowid > 0) {
        auto begin_itr = row_ptr.tbegin() + batch.base_rowid;
        auto end_itr = begin_itr + batch.Size() + 1;
        IncrementOffset(begin_itr, end_itr, batch.base_rowid);
      }
      dh::safe_cuda(hipMemcpy(data.Data() + data_offset, batch.data.data(),
                               sizeof(Entry) * batch.data.size(),
                               hipMemcpyHostToDevice));
      // Copy data
      data_offset += batch.data.size();
    }
  }
};

/**
 * \struct  DevicePredictionNode
 *
 * \brief Packed 16 byte representation of a tree node for use in device
 * prediction
 */

struct DevicePredictionNode {
  XGBOOST_DEVICE DevicePredictionNode()
      : fidx(-1), left_child_idx(-1), right_child_idx(-1) {}

  union NodeValue {
    float leaf_weight;
    float fvalue;
  };

  int fidx;
  int left_child_idx;
  int right_child_idx;
  NodeValue val;

  DevicePredictionNode(const RegTree::Node& n) {  // NOLINT
    this->left_child_idx = n.LeftChild();
    this->right_child_idx = n.RightChild();
    this->fidx = n.SplitIndex();
    if (n.DefaultLeft()) {
      fidx |= (1U << 31);
    }

    if (n.IsLeaf()) {
      this->val.leaf_weight = n.LeafValue();
    } else {
      this->val.fvalue = n.SplitCond();
    }
  }

  XGBOOST_DEVICE bool IsLeaf() const { return left_child_idx == -1; }

  XGBOOST_DEVICE int GetFidx() const { return fidx & ((1U << 31) - 1U); }

  XGBOOST_DEVICE bool MissingLeft() const { return (fidx >> 31) != 0; }

  XGBOOST_DEVICE int MissingIdx() const {
    if (MissingLeft()) {
      return this->left_child_idx;
    } else {
      return this->right_child_idx;
    }
  }

  XGBOOST_DEVICE float GetFvalue() const { return val.fvalue; }

  XGBOOST_DEVICE float GetWeight() const { return val.leaf_weight; }
};

struct ElementLoader {
  bool use_shared;
  size_t* d_row_ptr;
  Entry* d_data;
  int num_features;
  float* smem;

  __device__ ElementLoader(bool use_shared, size_t* row_ptr,
                           Entry* entry, int num_features,
                           float* smem, int num_rows)
      : use_shared(use_shared),
        d_row_ptr(row_ptr),
        d_data(entry),
        num_features(num_features),
        smem(smem) {
    // Copy instances
    if (use_shared) {
      bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
      int shared_elements = blockDim.x * num_features;
      dh::BlockFill(smem, shared_elements, nanf(""));
      __syncthreads();
      if (global_idx < num_rows) {
        bst_uint elem_begin = d_row_ptr[global_idx];
        bst_uint elem_end = d_row_ptr[global_idx + 1];
        for (bst_uint elem_idx = elem_begin; elem_idx < elem_end; elem_idx++) {
          Entry elem = d_data[elem_idx];
          smem[threadIdx.x * num_features + elem.index] = elem.fvalue;
        }
      }
      __syncthreads();
    }
  }
  __device__ float GetFvalue(int ridx, int fidx) {
    if (use_shared) {
      return smem[threadIdx.x * num_features + fidx];
    } else {
      // Binary search
      auto begin_ptr = d_data + d_row_ptr[ridx];
      auto end_ptr = d_data + d_row_ptr[ridx + 1];
      Entry* previous_middle = nullptr;
      while (end_ptr != begin_ptr) {
        auto middle = begin_ptr + (end_ptr - begin_ptr) / 2;
        if (middle == previous_middle) {
          break;
        } else {
          previous_middle = middle;
        }

        if (middle->index == fidx) {
          return middle->fvalue;
        } else if (middle->index < fidx) {
          begin_ptr = middle;
        } else {
          end_ptr = middle;
        }
      }
      // Value is missing
      return nanf("");
    }
  }
};

__device__ float GetLeafWeight(bst_uint ridx, const DevicePredictionNode* tree,
                               ElementLoader* loader) {
  DevicePredictionNode n = tree[0];
  while (!n.IsLeaf()) {
    float fvalue = loader->GetFvalue(ridx, n.GetFidx());
    // Missing value
    if (isnan(fvalue)) {
      n = tree[n.MissingIdx()];
    } else {
      if (fvalue < n.GetFvalue()) {
        n = tree[n.left_child_idx];
      } else {
        n = tree[n.right_child_idx];
      }
    }
  }
  return n.GetWeight();
}

template <int BLOCK_THREADS>
__global__ void PredictKernel(const DevicePredictionNode* d_nodes,
                              float* d_out_predictions, size_t* d_tree_segments,
                              int* d_tree_group, size_t* d_row_ptr,
                              Entry* d_data, size_t tree_begin,
                              size_t tree_end, size_t num_features,
                              size_t num_rows, bool use_shared, int num_group) {
  extern __shared__ float smem[];
  bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
  ElementLoader loader(use_shared, d_row_ptr, d_data, num_features, smem,
                       num_rows);
  if (global_idx >= num_rows) return;
  if (num_group == 1) {
    float sum = 0;
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      const DevicePredictionNode* d_tree =
          d_nodes + d_tree_segments[tree_idx - tree_begin];
      sum += GetLeafWeight(global_idx, d_tree, &loader);
    }
    d_out_predictions[global_idx] += sum;
  } else {
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      int tree_group = d_tree_group[tree_idx];
      const DevicePredictionNode* d_tree =
          d_nodes + d_tree_segments[tree_idx - tree_begin];
      bst_uint out_prediction_idx = global_idx * num_group + tree_group;
      d_out_predictions[out_prediction_idx] +=
          GetLeafWeight(global_idx, d_tree, &loader);
    }
  }
}

class GPUPredictor : public xgboost::Predictor {
 protected:
  struct DevicePredictionCacheEntry {
    std::shared_ptr<DMatrix> data;
    HostDeviceVector<bst_float> predictions;
  };

 private:
  void DevicePredictInternal(DMatrix* dmat,
                             HostDeviceVector<bst_float>* out_preds,
                             const gbm::GBTreeModel& model, size_t tree_begin,
                             size_t tree_end) {
    if (tree_end - tree_begin == 0) {
      return;
    }

    std::shared_ptr<DeviceMatrix> device_matrix;
    // Matrix is not in host cache, create a temporary matrix
    if (this->cache_.find(dmat) == this->cache_.end()) {
      device_matrix = std::shared_ptr<DeviceMatrix>(
          new DeviceMatrix(dmat, param.gpu_id, param.silent));
    } else {
      // Create this matrix on device if doesn't exist
      if (this->device_matrix_cache_.find(dmat) ==
          this->device_matrix_cache_.end()) {
        this->device_matrix_cache_.emplace(
            dmat, std::shared_ptr<DeviceMatrix>(
                      new DeviceMatrix(dmat, param.gpu_id, param.silent)));
      }
      device_matrix = device_matrix_cache_.find(dmat)->second;
    }

    dh::safe_cuda(hipSetDevice(param.gpu_id));
    CHECK_EQ(model.param.size_leaf_vector, 0);
    // Copy decision trees to device
    thrust::host_vector<size_t> h_tree_segments;
    h_tree_segments.reserve((tree_end - tree_begin) + 1);
    size_t sum = 0;
    h_tree_segments.push_back(sum);
    for (auto tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      sum += model.trees[tree_idx]->GetNodes().size();
      h_tree_segments.push_back(sum);
    }

    thrust::host_vector<DevicePredictionNode> h_nodes(h_tree_segments.back());
    for (auto tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      auto& src_nodes = model.trees[tree_idx]->GetNodes();
      std::copy(src_nodes.begin(), src_nodes.end(),
                h_nodes.begin() + h_tree_segments[tree_idx - tree_begin]);
    }

    nodes.resize(h_nodes.size());
    dh::safe_cuda(hipMemcpy(dh::Raw(nodes), h_nodes.data(),
                             sizeof(DevicePredictionNode) * h_nodes.size(),
                             hipMemcpyHostToDevice));
    tree_segments.resize(h_tree_segments.size());
    dh::safe_cuda(hipMemcpy(dh::Raw(tree_segments), h_tree_segments.data(),
                             sizeof(size_t) * h_tree_segments.size(),
                             hipMemcpyHostToDevice));
    tree_group.resize(model.tree_info.size());
    dh::safe_cuda(hipMemcpy(dh::Raw(tree_group), model.tree_info.data(),
                             sizeof(int) * model.tree_info.size(),
                             hipMemcpyHostToDevice));

    device_matrix->predictions.resize(out_preds->Size());
    auto& predictions = device_matrix->predictions;
    out_preds->GatherTo(predictions.data(),
                        predictions.data() + predictions.size());

    dh::safe_cuda(hipSetDevice(param.gpu_id));

    const int BLOCK_THREADS = 128;
    const int GRID_SIZE = static_cast<int>(
        dh::DivRoundUp(device_matrix->row_ptr.Size() - 1, BLOCK_THREADS));

    int shared_memory_bytes = static_cast<int>(
        sizeof(float) * device_matrix->p_mat->Info().num_col_ * BLOCK_THREADS);
    bool use_shared = true;
    if (shared_memory_bytes > max_shared_memory_bytes) {
      shared_memory_bytes = 0;
      use_shared = false;
    }

    PredictKernel<BLOCK_THREADS>
        <<<GRID_SIZE, BLOCK_THREADS, shared_memory_bytes>>>(
            dh::Raw(nodes), dh::Raw(device_matrix->predictions),
            dh::Raw(tree_segments), dh::Raw(tree_group),
            device_matrix->row_ptr.Data(), device_matrix->data.Data(),
            tree_begin, tree_end, device_matrix->p_mat->Info().num_col_,
            device_matrix->p_mat->Info().num_row_, use_shared,
            model.param.num_output_group);

    dh::safe_cuda(hipDeviceSynchronize());
    out_preds->ScatterFrom(predictions.data(),
                           predictions.data() + predictions.size());
  }

 public:
  GPUPredictor() : cpu_predictor(Predictor::Create("cpu_predictor")) {}

  void PredictBatch(DMatrix* dmat, HostDeviceVector<bst_float>* out_preds,
                    const gbm::GBTreeModel& model, int tree_begin,
                    unsigned ntree_limit = 0) override {
    if (this->PredictFromCache(dmat, out_preds, model, ntree_limit)) {
      return;
    }
    this->InitOutPredictions(dmat->Info(), out_preds, model);

    int tree_end = ntree_limit * model.param.num_output_group;

    if (ntree_limit == 0 || ntree_limit > model.trees.size()) {
      tree_end = static_cast<unsigned>(model.trees.size());
    }

    DevicePredictInternal(dmat, out_preds, model, tree_begin, tree_end);
  }

 protected:
  void InitOutPredictions(const MetaInfo& info,
                          HostDeviceVector<bst_float>* out_preds,
                          const gbm::GBTreeModel& model) const {
    size_t n = model.param.num_output_group * info.num_row_;
    const std::vector<bst_float>& base_margin = info.base_margin_;
    out_preds->Reshard(devices);
    out_preds->Resize(n);
    if (base_margin.size() != 0) {
      CHECK_EQ(out_preds->Size(), n);
      out_preds->Copy(base_margin);
    } else {
      out_preds->Fill(model.base_margin);
    }
  }

  bool PredictFromCache(DMatrix* dmat, HostDeviceVector<bst_float>* out_preds,
                        const gbm::GBTreeModel& model, unsigned ntree_limit) {
    if (ntree_limit == 0 ||
        ntree_limit * model.param.num_output_group >= model.trees.size()) {
      auto it = cache_.find(dmat);
      if (it != cache_.end()) {
        HostDeviceVector<bst_float>& y = it->second.predictions;
        if (y.Size() != 0) {
          out_preds->Reshard(devices);
          out_preds->Resize(y.Size());
          out_preds->Copy(&y);
          return true;
        }
      }
    }

    return false;
  }

  void UpdatePredictionCache(
      const gbm::GBTreeModel& model,
      std::vector<std::unique_ptr<TreeUpdater>>* updaters,
      int num_new_trees) override {
    auto old_ntree = model.trees.size() - num_new_trees;
    // update cache entry
    for (auto& kv : cache_) {
      PredictionCacheEntry& e = kv.second;
      DMatrix* dmat = kv.first;
      HostDeviceVector<bst_float>& predictions = e.predictions;

      if (predictions.Size() == 0) {
        this->InitOutPredictions(dmat->Info(), &predictions, model);
      }

      if (model.param.num_output_group == 1 && updaters->size() > 0 &&
          num_new_trees == 1 &&
          updaters->back()->UpdatePredictionCache(e.data.get(), &predictions)) {
        // do nothing
      } else {
        DevicePredictInternal(dmat, &predictions, model, old_ntree, model.trees.size());
      }
    }
  }

  void PredictInstance(const SparsePage::Inst& inst,
                       std::vector<bst_float>* out_preds,
                       const gbm::GBTreeModel& model, unsigned ntree_limit,
                       unsigned root_index) override {
    cpu_predictor->PredictInstance(inst, out_preds, model, root_index);
  }
  void PredictLeaf(DMatrix* p_fmat, std::vector<bst_float>* out_preds,
                   const gbm::GBTreeModel& model,
                   unsigned ntree_limit) override {
    cpu_predictor->PredictLeaf(p_fmat, out_preds, model, ntree_limit);
  }

  void PredictContribution(DMatrix* p_fmat,
                           std::vector<bst_float>* out_contribs,
                           const gbm::GBTreeModel& model, unsigned ntree_limit,
                           bool approximate, int condition,
                           unsigned condition_feature) override {
    cpu_predictor->PredictContribution(p_fmat, out_contribs, model, ntree_limit,
                                       approximate, condition,
                                       condition_feature);
  }

  void PredictInteractionContributions(DMatrix* p_fmat,
                                       std::vector<bst_float>* out_contribs,
                                       const gbm::GBTreeModel& model,
                                       unsigned ntree_limit,
                                       bool approximate) override {
    cpu_predictor->PredictInteractionContributions(p_fmat, out_contribs, model,
                                                   ntree_limit, approximate);
  }

  void Init(const std::vector<std::pair<std::string, std::string>>& cfg,
            const std::vector<std::shared_ptr<DMatrix>>& cache) override {
    Predictor::Init(cfg, cache);
    cpu_predictor->Init(cfg, cache);
    param.InitAllowUnknown(cfg);
    devices = GPUSet::Range(param.gpu_id, dh::NDevicesAll(param.n_gpus));
    max_shared_memory_bytes = dh::MaxSharedMemory(param.gpu_id);
  }

 private:
  GPUPredictionParam param;
  std::unique_ptr<Predictor> cpu_predictor;
  std::unordered_map<DMatrix*, std::shared_ptr<DeviceMatrix>>
      device_matrix_cache_;
  thrust::device_vector<DevicePredictionNode> nodes;
  thrust::device_vector<size_t> tree_segments;
  thrust::device_vector<int> tree_group;
  thrust::device_vector<bst_float> preds;
  GPUSet devices;
  size_t max_shared_memory_bytes;
};
XGBOOST_REGISTER_PREDICTOR(GPUPredictor, "gpu_predictor")
    .describe("Make predictions using GPU.")
    .set_body([]() { return new GPUPredictor(); });
}  // namespace predictor
}  // namespace xgboost
