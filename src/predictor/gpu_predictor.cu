#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2018 by Contributors
 */
#include <dmlc/parameter.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <xgboost/data.h>
#include <xgboost/predictor.h>
#include <xgboost/tree_model.h>
#include <xgboost/tree_updater.h>
#include <memory>
#include "../common/common.h"
#include "../common/device_helpers.cuh"
#include "../common/host_device_vector.h"

namespace xgboost {
namespace predictor {

DMLC_REGISTRY_FILE_TAG(gpu_predictor);

template <typename IterT>
void IncrementOffset(IterT begin_itr, IterT end_itr, size_t amount) {
  thrust::transform(begin_itr, end_itr, begin_itr,
                    [=] __device__(size_t elem) { return elem + amount; });
}

/**
 * \struct  DevicePredictionNode
 *
 * \brief Packed 16 byte representation of a tree node for use in device
 * prediction
 */
struct DevicePredictionNode {
  XGBOOST_DEVICE DevicePredictionNode()
      : fidx{-1}, left_child_idx{-1}, right_child_idx{-1} {}

  union NodeValue {
    float leaf_weight;
    float fvalue;
  };

  int fidx;
  int left_child_idx;
  int right_child_idx;
  NodeValue val;

  DevicePredictionNode(const RegTree::Node& n) {  // NOLINT
    static_assert(sizeof(DevicePredictionNode) == 16, "Size is not 16 bytes");
    this->left_child_idx = n.LeftChild();
    this->right_child_idx = n.RightChild();
    this->fidx = n.SplitIndex();
    if (n.DefaultLeft()) {
      fidx |= (1U << 31);
    }

    if (n.IsLeaf()) {
      this->val.leaf_weight = n.LeafValue();
    } else {
      this->val.fvalue = n.SplitCond();
    }
  }

  XGBOOST_DEVICE bool IsLeaf() const { return left_child_idx == -1; }

  XGBOOST_DEVICE int GetFidx() const { return fidx & ((1U << 31) - 1U); }

  XGBOOST_DEVICE bool MissingLeft() const { return (fidx >> 31) != 0; }

  XGBOOST_DEVICE int MissingIdx() const {
    if (MissingLeft()) {
      return this->left_child_idx;
    } else {
      return this->right_child_idx;
    }
  }

  XGBOOST_DEVICE float GetFvalue() const { return val.fvalue; }

  XGBOOST_DEVICE float GetWeight() const { return val.leaf_weight; }
};

struct ElementLoader {
  bool use_shared;
  common::Span<const size_t> d_row_ptr;
  common::Span<const Entry> d_data;
  int num_features;
  float* smem;
  size_t entry_start;

  __device__ ElementLoader(bool use_shared, common::Span<const size_t> row_ptr,
                           common::Span<const Entry> entry, int num_features,
                           float* smem, int num_rows, size_t entry_start)
      : use_shared(use_shared),
        d_row_ptr(row_ptr),
        d_data(entry),
        num_features(num_features),
        smem(smem),
        entry_start(entry_start) {
    // Copy instances
    if (use_shared) {
      bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
      int shared_elements = blockDim.x * num_features;
      dh::BlockFill(smem, shared_elements, nanf(""));
      __syncthreads();
      if (global_idx < num_rows) {
        bst_uint elem_begin = d_row_ptr[global_idx];
        bst_uint elem_end = d_row_ptr[global_idx + 1];
        for (bst_uint elem_idx = elem_begin; elem_idx < elem_end; elem_idx++) {
          Entry elem = d_data[elem_idx - entry_start];
          smem[threadIdx.x * num_features + elem.index] = elem.fvalue;
        }
      }
      __syncthreads();
    }
  }
  __device__ float GetFvalue(int ridx, int fidx) {
    if (use_shared) {
      return smem[threadIdx.x * num_features + fidx];
    } else {
      // Binary search
      auto begin_ptr = d_data.begin() + (d_row_ptr[ridx] - entry_start);
      auto end_ptr = d_data.begin() + (d_row_ptr[ridx + 1] - entry_start);
      common::Span<const Entry>::iterator previous_middle;
      while (end_ptr != begin_ptr) {
        auto middle = begin_ptr + (end_ptr - begin_ptr) / 2;
        if (middle == previous_middle) {
          break;
        } else {
          previous_middle = middle;
        }

        if (middle->index == fidx) {
          return middle->fvalue;
        } else if (middle->index < fidx) {
          begin_ptr = middle;
        } else {
          end_ptr = middle;
        }
      }
      // Value is missing
      return nanf("");
    }
  }
};

__device__ float GetLeafWeight(bst_uint ridx, const DevicePredictionNode* tree,
                               ElementLoader* loader) {
  DevicePredictionNode n = tree[0];
  while (!n.IsLeaf()) {
    float fvalue = loader->GetFvalue(ridx, n.GetFidx());
    // Missing value
    if (isnan(fvalue)) {
      n = tree[n.MissingIdx()];
    } else {
      if (fvalue < n.GetFvalue()) {
        n = tree[n.left_child_idx];
      } else {
        n = tree[n.right_child_idx];
      }
    }
  }
  return n.GetWeight();
}

template <int BLOCK_THREADS>
__global__ void PredictKernel(common::Span<const DevicePredictionNode> d_nodes,
                              common::Span<float> d_out_predictions,
                              common::Span<size_t> d_tree_segments,
                              common::Span<int> d_tree_group,
                              common::Span<const size_t> d_row_ptr,
                              common::Span<const Entry> d_data, size_t tree_begin,
                              size_t tree_end, size_t num_features,
                              size_t num_rows, size_t entry_start,
                              bool use_shared, int num_group) {
  extern __shared__ float smem[];
  bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
  ElementLoader loader(use_shared, d_row_ptr, d_data, num_features, smem,
                       num_rows, entry_start);
  if (global_idx >= num_rows) return;
  if (num_group == 1) {
    float sum = 0;
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      const DevicePredictionNode* d_tree =
          &d_nodes[d_tree_segments[tree_idx - tree_begin]];
      sum += GetLeafWeight(global_idx, d_tree, &loader);
    }
    d_out_predictions[global_idx] += sum;
  } else {
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      int tree_group = d_tree_group[tree_idx];
      const DevicePredictionNode* d_tree =
          &d_nodes[d_tree_segments[tree_idx - tree_begin]];
      bst_uint out_prediction_idx = global_idx * num_group + tree_group;
      d_out_predictions[out_prediction_idx] +=
          GetLeafWeight(global_idx, d_tree, &loader);
    }
  }
}

class GPUPredictor : public xgboost::Predictor {
 protected:
  struct DevicePredictionCacheEntry {
    std::shared_ptr<DMatrix> data;
    HostDeviceVector<bst_float> predictions;
  };

 private:
  void DeviceOffsets(const HostDeviceVector<size_t>& data,
                     size_t total_size,
                     std::vector<size_t>* out_offsets) {
    auto& offsets = *out_offsets;
    offsets.resize(devices_.Size() + 1);
    offsets[0] = 0;
#pragma omp parallel for schedule(static, 1) if (devices_.Size() > 1)
    for (int shard = 0; shard < devices_.Size(); ++shard) {
      int device = devices_.DeviceId(shard);
      auto data_span = data.DeviceSpan(device);
      dh::safe_cuda(hipSetDevice(device));
      if (data_span.size() == 0) {
        offsets[shard + 1] = total_size;
      } else {
        // copy the last element from every shard
        dh::safe_cuda(hipMemcpy(&offsets.at(shard + 1),
                                 &data_span[data_span.size()-1],
                                 sizeof(size_t), hipMemcpyDeviceToHost));
      }
    }
  }

  // This function populates the explicit offsets that can be used to create a window into the
  // underlying host vector. The window starts from the `batch_offset` and has a size of
  // `batch_size`, and is sharded across all the devices. Each shard is granular depending on
  // the number of output classes `n_classes`.
  void PredictionDeviceOffsets(size_t total_size, size_t batch_offset, size_t batch_size,
                               int n_classes, std::vector<size_t>* out_offsets) {
    auto& offsets = *out_offsets;
    size_t n_shards = devices_.Size();
    offsets.resize(n_shards + 2);
    size_t rows_per_shard = dh::DivRoundUp(batch_size, n_shards);
    for (size_t shard = 0; shard < devices_.Size(); ++shard) {
      size_t n_rows = std::min(batch_size, shard * rows_per_shard);
      offsets[shard] = batch_offset + n_rows * n_classes;
    }
    offsets[n_shards] = batch_offset + batch_size * n_classes;
    offsets[n_shards + 1] = total_size;
  }

  struct DeviceShard {
    DeviceShard() : device_{-1} {}

    void Init(int device) {
      this->device_ = device;
      max_shared_memory_bytes_ = dh::MaxSharedMemory(this->device_);
     }

    void InitModel(const gbm::GBTreeModel& model,
     const thrust::host_vector<size_t>& h_tree_segments,
     const thrust::host_vector<DevicePredictionNode>& h_nodes,
     size_t tree_begin, size_t tree_end) {
      dh::safe_cuda(hipSetDevice(device_));
      nodes_.resize(h_nodes.size());
      dh::safe_cuda(hipMemcpyAsync(dh::Raw(nodes_), h_nodes.data(),
                                    sizeof(DevicePredictionNode) * h_nodes.size(),
                                    hipMemcpyHostToDevice));
      tree_segments_.resize(h_tree_segments.size());
      dh::safe_cuda(hipMemcpyAsync(dh::Raw(tree_segments_), h_tree_segments.data(),
                                    sizeof(size_t) * h_tree_segments.size(),
                                    hipMemcpyHostToDevice));
      tree_group_.resize(model.tree_info.size());
      dh::safe_cuda(hipMemcpyAsync(dh::Raw(tree_group_), model.tree_info.data(),
                                    sizeof(int) * model.tree_info.size(),
                                    hipMemcpyHostToDevice));
      this->tree_begin_ = tree_begin;
      this->tree_end_ = tree_end;
      this->num_group_ = model.param.num_output_group;
    }

    void PredictInternal
    (const SparsePage& batch, const MetaInfo& info,
     HostDeviceVector<bst_float>* predictions) {
      if (predictions->DeviceSize(device_) == 0) { return; }
      dh::safe_cuda(hipSetDevice(device_));
      const int BLOCK_THREADS = 128;
      size_t num_rows = batch.offset.DeviceSize(device_) - 1;
      const int GRID_SIZE = static_cast<int>(dh::DivRoundUp(num_rows, BLOCK_THREADS));

      int shared_memory_bytes = static_cast<int>
        (sizeof(float) * info.num_col_ * BLOCK_THREADS);
      bool use_shared = true;
      if (shared_memory_bytes > max_shared_memory_bytes_) {
        shared_memory_bytes = 0;
        use_shared = false;
      }
      const auto& data_distr = batch.data.Distribution();
      size_t entry_start = data_distr.ShardStart(batch.data.Size(),
                                                 data_distr.Devices().Index(device_));

      PredictKernel<BLOCK_THREADS><<<GRID_SIZE, BLOCK_THREADS, shared_memory_bytes>>>
        (dh::ToSpan(nodes_), predictions->DeviceSpan(device_), dh::ToSpan(tree_segments_),
         dh::ToSpan(tree_group_), batch.offset.DeviceSpan(device_),
         batch.data.DeviceSpan(device_), this->tree_begin_, this->tree_end_, info.num_col_,
         num_rows, entry_start, use_shared, this->num_group_);
    }

   private:
    int device_;
    thrust::device_vector<DevicePredictionNode> nodes_;
    thrust::device_vector<size_t> tree_segments_;
    thrust::device_vector<int> tree_group_;
    size_t max_shared_memory_bytes_;
    size_t tree_begin_;
    size_t tree_end_;
    int num_group_;
  };

  void InitModel(const gbm::GBTreeModel& model, size_t tree_begin, size_t tree_end) {
    CHECK_EQ(model.param.size_leaf_vector, 0);
    // Copy decision trees to device
    thrust::host_vector<size_t> h_tree_segments;
    h_tree_segments.reserve((tree_end - tree_begin) + 1);
    size_t sum = 0;
    h_tree_segments.push_back(sum);
    for (auto tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      sum += model.trees.at(tree_idx)->GetNodes().size();
      h_tree_segments.push_back(sum);
    }

    thrust::host_vector<DevicePredictionNode> h_nodes(h_tree_segments.back());
    for (auto tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      auto& src_nodes = model.trees.at(tree_idx)->GetNodes();
      std::copy(src_nodes.begin(), src_nodes.end(),
                h_nodes.begin() + h_tree_segments[tree_idx - tree_begin]);
    }
    dh::ExecuteIndexShards(&shards_, [&](int idx, DeviceShard &shard) {
      shard.InitModel(model, h_tree_segments, h_nodes, tree_begin, tree_end);
    });
  }

  void DevicePredictInternal(DMatrix* dmat,
                             HostDeviceVector<bst_float>* out_preds,
                             const gbm::GBTreeModel& model, size_t tree_begin,
                             size_t tree_end) {
    if (tree_end - tree_begin == 0) { return; }
    monitor_.StartCuda("DevicePredictInternal");

    InitModel(model, tree_begin, tree_end);

    size_t batch_offset = 0;
    for (auto &batch : dmat->GetRowBatches()) {
      bool is_external_memory = batch.Size() < dmat->Info().num_row_;
      if (is_external_memory) {
        std::vector<size_t> out_preds_offsets;
        PredictionDeviceOffsets(out_preds->Size(), batch_offset, batch.Size(),
                                model.param.num_output_group, &out_preds_offsets);
        out_preds->Reshard(GPUDistribution::Explicit(devices_, out_preds_offsets));
      }

      batch.offset.Shard(GPUDistribution::Overlap(devices_, 1));
      std::vector<size_t> device_offsets;
      DeviceOffsets(batch.offset, batch.data.Size(), &device_offsets);
      batch.data.Reshard(GPUDistribution::Explicit(devices_, device_offsets));

      dh::ExecuteIndexShards(&shards_, [&](int idx, DeviceShard& shard) {
        shard.PredictInternal(batch, dmat->Info(), out_preds);
      });
      batch_offset += batch.Size() * model.param.num_output_group;
    }
    out_preds->Reshard(GPUDistribution::Granular(devices_, model.param.num_output_group));

    monitor_.StopCuda("DevicePredictInternal");
  }

 public:
  GPUPredictor()
      : cpu_predictor_(Predictor::Create("cpu_predictor", learner_param_)) {}

  void PredictBatch(DMatrix* dmat, HostDeviceVector<bst_float>* out_preds,
                    const gbm::GBTreeModel& model, int tree_begin,
                    unsigned ntree_limit = 0) override {
    GPUSet devices = GPUSet::All(learner_param_->gpu_id, learner_param_->n_gpus,
                                 dmat->Info().num_row_);
    CHECK_NE(devices.Size(), 0);
    ConfigureShards(devices);

    if (this->PredictFromCache(dmat, out_preds, model, ntree_limit)) {
      return;
    }
    this->InitOutPredictions(dmat->Info(), out_preds, model);

    int tree_end = ntree_limit * model.param.num_output_group;

    if (ntree_limit == 0 || ntree_limit > model.trees.size()) {
      tree_end = static_cast<unsigned>(model.trees.size());
    }

    DevicePredictInternal(dmat, out_preds, model, tree_begin, tree_end);
  }

 protected:
  void InitOutPredictions(const MetaInfo& info,
                          HostDeviceVector<bst_float>* out_preds,
                          const gbm::GBTreeModel& model) const {
    size_t n_classes = model.param.num_output_group;
    size_t n = n_classes * info.num_row_;
    const HostDeviceVector<bst_float>& base_margin = info.base_margin_;
    out_preds->Shard(GPUDistribution::Granular(devices_, n_classes));
    out_preds->Resize(n);
    if (base_margin.Size() != 0) {
      CHECK_EQ(out_preds->Size(), n);
      out_preds->Copy(base_margin);
    } else {
      out_preds->Fill(model.base_margin);
    }
  }

  bool PredictFromCache(DMatrix* dmat, HostDeviceVector<bst_float>* out_preds,
                        const gbm::GBTreeModel& model, unsigned ntree_limit) {
    if (ntree_limit == 0 ||
        ntree_limit * model.param.num_output_group >= model.trees.size()) {
      auto it = cache_.find(dmat);
      if (it != cache_.end()) {
        const HostDeviceVector<bst_float>& y = it->second.predictions;
        if (y.Size() != 0) {
          monitor_.StartCuda("PredictFromCache");
          out_preds->Shard(y.Distribution());
          out_preds->Resize(y.Size());
          out_preds->Copy(y);
          monitor_.StopCuda("PredictFromCache");
          return true;
        }
      }
    }
    return false;
  }

  void UpdatePredictionCache(
      const gbm::GBTreeModel& model,
      std::vector<std::unique_ptr<TreeUpdater>>* updaters,
      int num_new_trees) override {
    auto old_ntree = model.trees.size() - num_new_trees;
    // update cache entry
    for (auto& kv : cache_) {
      PredictionCacheEntry& e = kv.second;
      DMatrix* dmat = kv.first;
      HostDeviceVector<bst_float>& predictions = e.predictions;

      if (predictions.Size() == 0) {
        this->InitOutPredictions(dmat->Info(), &predictions, model);
      }

      if (model.param.num_output_group == 1 && updaters->size() > 0 &&
          num_new_trees == 1 &&
          updaters->back()->UpdatePredictionCache(e.data.get(), &predictions)) {
        // do nothing
      } else {
        DevicePredictInternal(dmat, &predictions, model, old_ntree, model.trees.size());
      }
    }
  }

  void PredictInstance(const SparsePage::Inst& inst,
                       std::vector<bst_float>* out_preds,
                       const gbm::GBTreeModel& model, unsigned ntree_limit,
                       unsigned root_index) override {
    cpu_predictor_->PredictInstance(inst, out_preds, model, root_index);
  }
  void PredictLeaf(DMatrix* p_fmat, std::vector<bst_float>* out_preds,
                   const gbm::GBTreeModel& model,
                   unsigned ntree_limit) override {
    cpu_predictor_->PredictLeaf(p_fmat, out_preds, model, ntree_limit);
  }

  void PredictContribution(DMatrix* p_fmat,
                           std::vector<bst_float>* out_contribs,
                           const gbm::GBTreeModel& model, unsigned ntree_limit,
                           bool approximate, int condition,
                           unsigned condition_feature) override {
    cpu_predictor_->PredictContribution(p_fmat, out_contribs, model, ntree_limit,
                                       approximate, condition,
                                       condition_feature);
  }

  void PredictInteractionContributions(DMatrix* p_fmat,
                                       std::vector<bst_float>* out_contribs,
                                       const gbm::GBTreeModel& model,
                                       unsigned ntree_limit,
                                       bool approximate) override {
    cpu_predictor_->PredictInteractionContributions(p_fmat, out_contribs, model,
                                                   ntree_limit, approximate);
  }

  void Init(const std::vector<std::pair<std::string, std::string>>& cfg,
            const std::vector<std::shared_ptr<DMatrix>>& cache) override {
    Predictor::Init(cfg, cache);
    cpu_predictor_->Init(cfg, cache);

    GPUSet devices = GPUSet::All(learner_param_->gpu_id, learner_param_->n_gpus);
    ConfigureShards(devices);
  }

 private:
  /*! \brief Re configure shards when GPUSet is changed. */
  void ConfigureShards(GPUSet devices) {
    if (devices_ == devices) return;

    devices_ = devices;
    shards_.clear();
    shards_.resize(devices_.Size());
    dh::ExecuteIndexShards(&shards_, [=](size_t i, DeviceShard& shard){
        shard.Init(devices_.DeviceId(i));
      });
  }

  std::unique_ptr<Predictor> cpu_predictor_;
  std::vector<DeviceShard> shards_;
  GPUSet devices_;
  common::Monitor monitor_;
};

XGBOOST_REGISTER_PREDICTOR(GPUPredictor, "gpu_predictor")
    .describe("Make predictions using GPU.")
    .set_body([]() { return new GPUPredictor(); });

}  // namespace predictor
}  // namespace xgboost
