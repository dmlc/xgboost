#include "hip/hip_runtime.h"
/*!
 * Copyright by Contributors 2017
 */
#include <dmlc/parameter.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <xgboost/data.h>
#include <xgboost/predictor.h>
#include <xgboost/tree_model.h>
#include <xgboost/tree_updater.h>
#include <memory>
#include "../common/common.h"
#include "../common/device_helpers.cuh"
#include "../common/host_device_vector.h"

namespace xgboost {
namespace predictor {

DMLC_REGISTRY_FILE_TAG(gpu_predictor);

/*! \brief prediction parameters */
struct GPUPredictionParam : public dmlc::Parameter<GPUPredictionParam> {
  int gpu_id;
  int n_gpus;
  bool silent;
  // declare parameters
  DMLC_DECLARE_PARAMETER(GPUPredictionParam) {
    DMLC_DECLARE_FIELD(gpu_id).set_lower_bound(0).set_default(0).describe(
        "Device ordinal for GPU prediction.");
    DMLC_DECLARE_FIELD(n_gpus).set_lower_bound(-1).set_default(1).describe(
        "Number of devices to use for prediction.");
    DMLC_DECLARE_FIELD(silent).set_default(false).describe(
        "Do not print information during trainig.");
  }
};
DMLC_REGISTER_PARAMETER(GPUPredictionParam);

template <typename IterT>
void IncrementOffset(IterT begin_itr, IterT end_itr, size_t amount) {
  thrust::transform(begin_itr, end_itr, begin_itr,
                    [=] __device__(size_t elem) { return elem + amount; });
}

/**
 * \struct  DevicePredictionNode
 *
 * \brief Packed 16 byte representation of a tree node for use in device
 * prediction
 */
struct DevicePredictionNode {
  XGBOOST_DEVICE DevicePredictionNode()
      : fidx(-1), left_child_idx(-1), right_child_idx(-1) {}

  union NodeValue {
    float leaf_weight;
    float fvalue;
  };

  int fidx;
  int left_child_idx;
  int right_child_idx;
  NodeValue val;

  DevicePredictionNode(const RegTree::Node& n) {  // NOLINT
    static_assert(sizeof(DevicePredictionNode) == 16, "Size is not 16 bytes");
    this->left_child_idx = n.LeftChild();
    this->right_child_idx = n.RightChild();
    this->fidx = n.SplitIndex();
    if (n.DefaultLeft()) {
      fidx |= (1U << 31);
    }

    if (n.IsLeaf()) {
      this->val.leaf_weight = n.LeafValue();
    } else {
      this->val.fvalue = n.SplitCond();
    }
  }

  XGBOOST_DEVICE bool IsLeaf() const { return left_child_idx == -1; }

  XGBOOST_DEVICE int GetFidx() const { return fidx & ((1U << 31) - 1U); }

  XGBOOST_DEVICE bool MissingLeft() const { return (fidx >> 31) != 0; }

  XGBOOST_DEVICE int MissingIdx() const {
    if (MissingLeft()) {
      return this->left_child_idx;
    } else {
      return this->right_child_idx;
    }
  }

  XGBOOST_DEVICE float GetFvalue() const { return val.fvalue; }

  XGBOOST_DEVICE float GetWeight() const { return val.leaf_weight; }
};

struct ElementLoader {
  bool use_shared;
  common::Span<const size_t> d_row_ptr;
  common::Span<const Entry> d_data;
  int num_features;
  float* smem;
  size_t entry_start;

  __device__ ElementLoader(bool use_shared, common::Span<const size_t> row_ptr,
                           common::Span<const Entry> entry, int num_features,
                           float* smem, int num_rows, size_t entry_start)
      : use_shared(use_shared),
        d_row_ptr(row_ptr),
        d_data(entry),
        num_features(num_features),
        smem(smem),
        entry_start(entry_start) {
    // Copy instances
    if (use_shared) {
      bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
      int shared_elements = blockDim.x * num_features;
      dh::BlockFill(smem, shared_elements, nanf(""));
      __syncthreads();
      if (global_idx < num_rows) {
        bst_uint elem_begin = d_row_ptr[global_idx];
        bst_uint elem_end = d_row_ptr[global_idx + 1];
        for (bst_uint elem_idx = elem_begin; elem_idx < elem_end; elem_idx++) {
          Entry elem = d_data[elem_idx - entry_start];
          smem[threadIdx.x * num_features + elem.index] = elem.fvalue;
        }
      }
      __syncthreads();
    }
  }
  __device__ float GetFvalue(int ridx, int fidx) {
    if (use_shared) {
      return smem[threadIdx.x * num_features + fidx];
    } else {
      // Binary search
      auto begin_ptr = d_data.begin() + (d_row_ptr[ridx] - entry_start);
      auto end_ptr = d_data.begin() + (d_row_ptr[ridx + 1] - entry_start);
      common::Span<const Entry>::iterator previous_middle;
      while (end_ptr != begin_ptr) {
        auto middle = begin_ptr + (end_ptr - begin_ptr) / 2;
        if (middle == previous_middle) {
          break;
        } else {
          previous_middle = middle;
        }

        if (middle->index == fidx) {
          return middle->fvalue;
        } else if (middle->index < fidx) {
          begin_ptr = middle;
        } else {
          end_ptr = middle;
        }
      }
      // Value is missing
      return nanf("");
    }
  }
};

__device__ float GetLeafWeight(bst_uint ridx, const DevicePredictionNode* tree,
                               ElementLoader* loader) {
  DevicePredictionNode n = tree[0];
  while (!n.IsLeaf()) {
    float fvalue = loader->GetFvalue(ridx, n.GetFidx());
    // Missing value
    if (isnan(fvalue)) {
      n = tree[n.MissingIdx()];
    } else {
      if (fvalue < n.GetFvalue()) {
        n = tree[n.left_child_idx];
      } else {
        n = tree[n.right_child_idx];
      }
    }
  }
  return n.GetWeight();
}

template <int BLOCK_THREADS>
__global__ void PredictKernel(common::Span<const DevicePredictionNode> d_nodes,
                              common::Span<float> d_out_predictions,
                              common::Span<size_t> d_tree_segments,
                              common::Span<int> d_tree_group,
                              common::Span<const size_t> d_row_ptr,
                              common::Span<const Entry> d_data, size_t tree_begin,
                              size_t tree_end, size_t num_features,
                              size_t num_rows, size_t entry_start,
                              bool use_shared, int num_group) {
  extern __shared__ float smem[];
  bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
  ElementLoader loader(use_shared, d_row_ptr, d_data, num_features, smem,
                       num_rows, entry_start);
  if (global_idx >= num_rows) return;
  if (num_group == 1) {
    float sum = 0;
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      const DevicePredictionNode* d_tree =
          &d_nodes[d_tree_segments[tree_idx - tree_begin]];
      sum += GetLeafWeight(global_idx, d_tree, &loader);
    }
    d_out_predictions[global_idx] += sum;
  } else {
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      int tree_group = d_tree_group[tree_idx];
      const DevicePredictionNode* d_tree =
          &d_nodes[d_tree_segments[tree_idx - tree_begin]];
      bst_uint out_prediction_idx = global_idx * num_group + tree_group;
      d_out_predictions[out_prediction_idx] +=
          GetLeafWeight(global_idx, d_tree, &loader);
    }
  }
}

class GPUPredictor : public xgboost::Predictor {
 protected:
  struct DevicePredictionCacheEntry {
    std::shared_ptr<DMatrix> data;
    HostDeviceVector<bst_float> predictions;
  };

 private:
  void DeviceOffsets(const HostDeviceVector<size_t>& data, std::vector<size_t>* out_offsets) {
    auto& offsets = *out_offsets;
    offsets.resize(devices_.Size() + 1);
    offsets[0] = 0;
#pragma omp parallel for schedule(static, 1) if (devices_.Size() > 1)
    for (int shard = 0; shard < devices_.Size(); ++shard) {
      int device = devices_[shard];
      auto data_span = data.DeviceSpan(device);
      dh::safe_cuda(hipSetDevice(device));
      // copy the last element from every shard
      dh::safe_cuda(hipMemcpy(&offsets.at(shard + 1),
                               &data_span[data_span.size()-1],
                               sizeof(size_t), hipMemcpyDeviceToHost));
    }
  }

  struct DeviceShard {
    DeviceShard() : device_(-1) {}
    void Init(int device) {
      this->device_ = device;
      max_shared_memory_bytes = dh::MaxSharedMemory(this->device_);
     }
    void PredictInternal
    (const SparsePage& batch, const MetaInfo& info,
     HostDeviceVector<bst_float>* predictions,
     const gbm::GBTreeModel& model,
     const thrust::host_vector<size_t>& h_tree_segments,
     const thrust::host_vector<DevicePredictionNode>& h_nodes,
     size_t tree_begin, size_t tree_end) {
      dh::safe_cuda(hipSetDevice(device_));
      nodes.resize(h_nodes.size());
      dh::safe_cuda(hipMemcpy(dh::Raw(nodes), h_nodes.data(),
                               sizeof(DevicePredictionNode) * h_nodes.size(),
                               hipMemcpyHostToDevice));
      tree_segments.resize(h_tree_segments.size());

      dh::safe_cuda(hipMemcpy(dh::Raw(tree_segments), h_tree_segments.data(),
                               sizeof(size_t) * h_tree_segments.size(),
                               hipMemcpyHostToDevice));
      tree_group.resize(model.tree_info.size());

      dh::safe_cuda(hipMemcpy(dh::Raw(tree_group), model.tree_info.data(),
                               sizeof(int) * model.tree_info.size(),
                               hipMemcpyHostToDevice));

      const int BLOCK_THREADS = 128;
      size_t num_rows = batch.offset.DeviceSize(device_) - 1;

      const int GRID_SIZE = static_cast<int>(dh::DivRoundUp(num_rows, BLOCK_THREADS));

      int shared_memory_bytes = static_cast<int>
        (sizeof(float) * info.num_col_ * BLOCK_THREADS);
      bool use_shared = true;
      if (shared_memory_bytes > max_shared_memory_bytes) {
        shared_memory_bytes = 0;
        use_shared = false;
      }
      const auto& data_distr = batch.data.Distribution();
      int index = data_distr.Devices().Index(device_);
      size_t entry_start = data_distr.ShardStart(batch.data.Size(), index);

      PredictKernel<BLOCK_THREADS><<<GRID_SIZE, BLOCK_THREADS, shared_memory_bytes>>>
        (dh::ToSpan(nodes), predictions->DeviceSpan(device_), dh::ToSpan(tree_segments),
         dh::ToSpan(tree_group), batch.offset.DeviceSpan(device_),
         batch.data.DeviceSpan(device_), tree_begin, tree_end, info.num_col_,
         num_rows, entry_start, use_shared, model.param.num_output_group);

      dh::safe_cuda(hipDeviceSynchronize());
    }

    int device_;
    thrust::device_vector<DevicePredictionNode> nodes;
    thrust::device_vector<size_t> tree_segments;
    thrust::device_vector<int> tree_group;
    size_t max_shared_memory_bytes;
  };

  void DevicePredictInternal(DMatrix* dmat,
                             HostDeviceVector<bst_float>* out_preds,
                             const gbm::GBTreeModel& model, size_t tree_begin,
                             size_t tree_end) {
    if (tree_end - tree_begin == 0) { return; }

    CHECK_EQ(model.param.size_leaf_vector, 0);
    // Copy decision trees to device
    thrust::host_vector<size_t> h_tree_segments;
    h_tree_segments.reserve((tree_end - tree_begin) + 1);
    size_t sum = 0;
    h_tree_segments.push_back(sum);
    for (auto tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      sum += model.trees.at(tree_idx)->GetNodes().size();
      h_tree_segments.push_back(sum);
    }

    thrust::host_vector<DevicePredictionNode> h_nodes(h_tree_segments.back());
    for (auto tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      auto& src_nodes = model.trees.at(tree_idx)->GetNodes();
      std::copy(src_nodes.begin(), src_nodes.end(),
                h_nodes.begin() + h_tree_segments[tree_idx - tree_begin]);
    }

    size_t i_batch = 0;

    for (const auto &batch : dmat->GetRowBatches()) {
      CHECK_EQ(i_batch, 0) << "External memory not supported";
      size_t n_rows = batch.offset.Size() - 1;
      // out_preds have been resharded and resized in InitOutPredictions()
      batch.offset.Reshard(GPUDistribution::Overlap(devices_, 1));
      std::vector<size_t> device_offsets;
      DeviceOffsets(batch.offset, &device_offsets);
      batch.data.Reshard(GPUDistribution::Explicit(devices_, device_offsets));
      dh::ExecuteShards(&shards, [&](DeviceShard& shard){
          shard.PredictInternal(batch, dmat->Info(), out_preds, model, h_tree_segments,
                                h_nodes, tree_begin, tree_end);
        });
      i_batch++;
    }
  }

 public:
  GPUPredictor() : cpu_predictor(Predictor::Create("cpu_predictor")) {}

  void PredictBatch(DMatrix* dmat, HostDeviceVector<bst_float>* out_preds,
                    const gbm::GBTreeModel& model, int tree_begin,
                    unsigned ntree_limit = 0) override {
    GPUSet devices = GPUSet::All(
        param.n_gpus, dmat->Info().num_row_).Normalised(param.gpu_id);
    ConfigureShards(devices);

    if (this->PredictFromCache(dmat, out_preds, model, ntree_limit)) {
      return;
    }
    this->InitOutPredictions(dmat->Info(), out_preds, model);

    int tree_end = ntree_limit * model.param.num_output_group;

    if (ntree_limit == 0 || ntree_limit > model.trees.size()) {
      tree_end = static_cast<unsigned>(model.trees.size());
    }

    DevicePredictInternal(dmat, out_preds, model, tree_begin, tree_end);
  }

 protected:
  void InitOutPredictions(const MetaInfo& info,
                          HostDeviceVector<bst_float>* out_preds,
                          const gbm::GBTreeModel& model) const {
    size_t n_classes = model.param.num_output_group;
    size_t n = n_classes * info.num_row_;
    const HostDeviceVector<bst_float>& base_margin = info.base_margin_;
    out_preds->Reshard(GPUDistribution::Granular(devices_, n_classes));
    out_preds->Resize(n);
    if (base_margin.Size() != 0) {
      CHECK_EQ(out_preds->Size(), n);
      out_preds->Copy(base_margin);
    } else {
      out_preds->Fill(model.base_margin);
    }
  }

  bool PredictFromCache(DMatrix* dmat, HostDeviceVector<bst_float>* out_preds,
                        const gbm::GBTreeModel& model, unsigned ntree_limit) {
    if (ntree_limit == 0 ||
        ntree_limit * model.param.num_output_group >= model.trees.size()) {
      auto it = cache_.find(dmat);
      if (it != cache_.end()) {
        const HostDeviceVector<bst_float>& y = it->second.predictions;
        if (y.Size() != 0) {
          out_preds->Reshard(y.Distribution());
          out_preds->Resize(y.Size());
          out_preds->Copy(y);
          return true;
        }
      }
    }
    return false;
  }

  void UpdatePredictionCache(
      const gbm::GBTreeModel& model,
      std::vector<std::unique_ptr<TreeUpdater>>* updaters,
      int num_new_trees) override {
    auto old_ntree = model.trees.size() - num_new_trees;
    // update cache entry
    for (auto& kv : cache_) {
      PredictionCacheEntry& e = kv.second;
      DMatrix* dmat = kv.first;
      HostDeviceVector<bst_float>& predictions = e.predictions;

      if (predictions.Size() == 0) {
        this->InitOutPredictions(dmat->Info(), &predictions, model);
      }

      if (model.param.num_output_group == 1 && updaters->size() > 0 &&
          num_new_trees == 1 &&
          updaters->back()->UpdatePredictionCache(e.data.get(), &predictions)) {
        // do nothing
      } else {
        DevicePredictInternal(dmat, &predictions, model, old_ntree, model.trees.size());
      }
    }
  }

  void PredictInstance(const SparsePage::Inst& inst,
                       std::vector<bst_float>* out_preds,
                       const gbm::GBTreeModel& model, unsigned ntree_limit,
                       unsigned root_index) override {
    cpu_predictor->PredictInstance(inst, out_preds, model, root_index);
  }
  void PredictLeaf(DMatrix* p_fmat, std::vector<bst_float>* out_preds,
                   const gbm::GBTreeModel& model,
                   unsigned ntree_limit) override {
    cpu_predictor->PredictLeaf(p_fmat, out_preds, model, ntree_limit);
  }

  void PredictContribution(DMatrix* p_fmat,
                           std::vector<bst_float>* out_contribs,
                           const gbm::GBTreeModel& model, unsigned ntree_limit,
                           bool approximate, int condition,
                           unsigned condition_feature) override {
    cpu_predictor->PredictContribution(p_fmat, out_contribs, model, ntree_limit,
                                       approximate, condition,
                                       condition_feature);
  }

  void PredictInteractionContributions(DMatrix* p_fmat,
                                       std::vector<bst_float>* out_contribs,
                                       const gbm::GBTreeModel& model,
                                       unsigned ntree_limit,
                                       bool approximate) override {
    cpu_predictor->PredictInteractionContributions(p_fmat, out_contribs, model,
                                                   ntree_limit, approximate);
  }

  void Init(const std::vector<std::pair<std::string, std::string>>& cfg,
            const std::vector<std::shared_ptr<DMatrix>>& cache) override {
    Predictor::Init(cfg, cache);
    cpu_predictor->Init(cfg, cache);
    param.InitAllowUnknown(cfg);

    GPUSet devices = GPUSet::All(param.n_gpus).Normalised(param.gpu_id);
    ConfigureShards(devices);
  }

 private:
  /*! \brief Re configure shards when GPUSet is changed. */
  void ConfigureShards(GPUSet devices) {
    if (devices_ == devices) return;

    devices_ = devices;
    shards.clear();
    shards.resize(devices_.Size());
    dh::ExecuteIndexShards(&shards, [=](size_t i, DeviceShard& shard){
        shard.Init(devices_[i]);
      });
  }

  GPUPredictionParam param;
  std::unique_ptr<Predictor> cpu_predictor;
  std::vector<DeviceShard> shards;
  GPUSet devices_;
};

XGBOOST_REGISTER_PREDICTOR(GPUPredictor, "gpu_predictor")
    .describe("Make predictions using GPU.")
    .set_body([]() { return new GPUPredictor(); });

}  // namespace predictor
}  // namespace xgboost
