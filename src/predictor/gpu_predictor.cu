#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2018 by Contributors
 */
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <memory>

#include "xgboost/parameter.h"
#include "xgboost/data.h"
#include "xgboost/predictor.h"
#include "xgboost/tree_model.h"
#include "xgboost/tree_updater.h"
#include "xgboost/host_device_vector.h"

#include "../gbm/gbtree_model.h"
#include "../common/common.h"
#include "../common/device_helpers.cuh"

namespace xgboost {
namespace predictor {

DMLC_REGISTRY_FILE_TAG(gpu_predictor);

/**
 * \struct  DevicePredictionNode
 *
 * \brief Packed 16 byte representation of a tree node for use in device
 * prediction
 */
struct DevicePredictionNode {
  XGBOOST_DEVICE DevicePredictionNode()
      : fidx{-1}, left_child_idx{-1}, right_child_idx{-1} {}

  union NodeValue {
    float leaf_weight;
    float fvalue;
  };

  int fidx;
  int left_child_idx;
  int right_child_idx;
  NodeValue val{};

  DevicePredictionNode(const RegTree::Node& n) {  // NOLINT
    static_assert(sizeof(DevicePredictionNode) == 16, "Size is not 16 bytes");
    this->left_child_idx = n.LeftChild();
    this->right_child_idx = n.RightChild();
    this->fidx = n.SplitIndex();
    if (n.DefaultLeft()) {
      fidx |= (1U << 31);
    }

    if (n.IsLeaf()) {
      this->val.leaf_weight = n.LeafValue();
    } else {
      this->val.fvalue = n.SplitCond();
    }
  }

  XGBOOST_DEVICE bool IsLeaf() const { return left_child_idx == -1; }

  XGBOOST_DEVICE int GetFidx() const { return fidx & ((1U << 31) - 1U); }

  XGBOOST_DEVICE bool MissingLeft() const { return (fidx >> 31) != 0; }

  XGBOOST_DEVICE int MissingIdx() const {
    if (MissingLeft()) {
      return this->left_child_idx;
    } else {
      return this->right_child_idx;
    }
  }

  XGBOOST_DEVICE float GetFvalue() const { return val.fvalue; }

  XGBOOST_DEVICE float GetWeight() const { return val.leaf_weight; }
};

struct ElementLoader {
  bool use_shared;
  common::Span<const bst_row_t> d_row_ptr;
  common::Span<const Entry> d_data;
  int num_features;
  float* smem;
  size_t entry_start;

  __device__ ElementLoader(bool use_shared, common::Span<const bst_row_t> row_ptr,
                           common::Span<const Entry> entry, int num_features,
                           float* smem, int num_rows, size_t entry_start)
      : use_shared(use_shared),
        d_row_ptr(row_ptr),
        d_data(entry),
        num_features(num_features),
        smem(smem),
        entry_start(entry_start) {
    // Copy instances
    if (use_shared) {
      bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
      int shared_elements = blockDim.x * num_features;
      dh::BlockFill(smem, shared_elements, nanf(""));
      __syncthreads();
      if (global_idx < num_rows) {
        bst_uint elem_begin = d_row_ptr[global_idx];
        bst_uint elem_end = d_row_ptr[global_idx + 1];
        for (bst_uint elem_idx = elem_begin; elem_idx < elem_end; elem_idx++) {
          Entry elem = d_data[elem_idx - entry_start];
          smem[threadIdx.x * num_features + elem.index] = elem.fvalue;
        }
      }
      __syncthreads();
    }
  }
  __device__ float GetFvalue(int ridx, int fidx) {
    if (use_shared) {
      return smem[threadIdx.x * num_features + fidx];
    } else {
      // Binary search
      auto begin_ptr = d_data.begin() + (d_row_ptr[ridx] - entry_start);
      auto end_ptr = d_data.begin() + (d_row_ptr[ridx + 1] - entry_start);
      common::Span<const Entry>::iterator previous_middle;
      while (end_ptr != begin_ptr) {
        auto middle = begin_ptr + (end_ptr - begin_ptr) / 2;
        if (middle == previous_middle) {
          break;
        } else {
          previous_middle = middle;
        }

        if (middle->index == fidx) {
          return middle->fvalue;
        } else if (middle->index < fidx) {
          begin_ptr = middle;
        } else {
          end_ptr = middle;
        }
      }
      // Value is missing
      return nanf("");
    }
  }
};

__device__ float GetLeafWeight(bst_uint ridx, const DevicePredictionNode* tree,
                               ElementLoader* loader) {
  DevicePredictionNode n = tree[0];
  while (!n.IsLeaf()) {
    float fvalue = loader->GetFvalue(ridx, n.GetFidx());
    // Missing value
    if (isnan(fvalue)) {
      n = tree[n.MissingIdx()];
    } else {
      if (fvalue < n.GetFvalue()) {
        n = tree[n.left_child_idx];
      } else {
        n = tree[n.right_child_idx];
      }
    }
  }
  return n.GetWeight();
}

template <int BLOCK_THREADS>
__global__ void PredictKernel(common::Span<const DevicePredictionNode> d_nodes,
                              common::Span<float> d_out_predictions,
                              common::Span<size_t> d_tree_segments,
                              common::Span<int> d_tree_group,
                              common::Span<const bst_row_t> d_row_ptr,
                              common::Span<const Entry> d_data, size_t tree_begin,
                              size_t tree_end, size_t num_features,
                              size_t num_rows, size_t entry_start,
                              bool use_shared, int num_group) {
  extern __shared__ float smem[];
  bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
  ElementLoader loader(use_shared, d_row_ptr, d_data, num_features, smem,
                       num_rows, entry_start);
  if (global_idx >= num_rows) return;
  if (num_group == 1) {
    float sum = 0;
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      const DevicePredictionNode* d_tree =
          &d_nodes[d_tree_segments[tree_idx - tree_begin]];
      sum += GetLeafWeight(global_idx, d_tree, &loader);
    }
    d_out_predictions[global_idx] += sum;
  } else {
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      int tree_group = d_tree_group[tree_idx];
      const DevicePredictionNode* d_tree =
          &d_nodes[d_tree_segments[tree_idx - tree_begin]];
      bst_uint out_prediction_idx = global_idx * num_group + tree_group;
      d_out_predictions[out_prediction_idx] +=
          GetLeafWeight(global_idx, d_tree, &loader);
    }
  }
}

class GPUPredictor : public xgboost::Predictor {
 private:
  void InitModel(const gbm::GBTreeModel& model,
   const thrust::host_vector<size_t>& h_tree_segments,
   const thrust::host_vector<DevicePredictionNode>& h_nodes,
   size_t tree_begin, size_t tree_end) {
    dh::safe_cuda(hipSetDevice(generic_param_->gpu_id));
    nodes_.resize(h_nodes.size());
    dh::safe_cuda(hipMemcpyAsync(nodes_.data().get(), h_nodes.data(),
                                  sizeof(DevicePredictionNode) * h_nodes.size(),
                                  hipMemcpyHostToDevice));
    tree_segments_.resize(h_tree_segments.size());
    dh::safe_cuda(hipMemcpyAsync(tree_segments_.data().get(), h_tree_segments.data(),
                                  sizeof(size_t) * h_tree_segments.size(),
                                  hipMemcpyHostToDevice));
    tree_group_.resize(model.tree_info.size());
    dh::safe_cuda(hipMemcpyAsync(tree_group_.data().get(), model.tree_info.data(),
                                  sizeof(int) * model.tree_info.size(),
                                  hipMemcpyHostToDevice));
    this->tree_begin_ = tree_begin;
    this->tree_end_ = tree_end;
    this->num_group_ = model.learner_model_param_->num_output_group;
  }

  void PredictInternal(const SparsePage& batch,
                       size_t num_features,
                       HostDeviceVector<bst_float>* predictions,
                       size_t batch_offset) {
    dh::safe_cuda(hipSetDevice(generic_param_->gpu_id));
    batch.data.SetDevice(generic_param_->gpu_id);
    batch.offset.SetDevice(generic_param_->gpu_id);
    predictions->SetDevice(generic_param_->gpu_id);

    const uint32_t BLOCK_THREADS = 128;
    size_t num_rows = batch.Size();
    auto GRID_SIZE = static_cast<uint32_t>(common::DivRoundUp(num_rows, BLOCK_THREADS));

    auto shared_memory_bytes =
        static_cast<size_t>(sizeof(float) * num_features * BLOCK_THREADS);
    bool use_shared = true;
    if (shared_memory_bytes > max_shared_memory_bytes_) {
      shared_memory_bytes = 0;
      use_shared = false;
    }
    size_t entry_start = 0;

    dh::LaunchKernel {GRID_SIZE, BLOCK_THREADS, shared_memory_bytes} (
        PredictKernel<BLOCK_THREADS>,
        dh::ToSpan(nodes_), predictions->DeviceSpan().subspan(batch_offset),
        dh::ToSpan(tree_segments_), dh::ToSpan(tree_group_), batch.offset.DeviceSpan(),
        batch.data.DeviceSpan(), this->tree_begin_, this->tree_end_, num_features, num_rows,
        entry_start, use_shared, this->num_group_);
  }

  void InitModel(const gbm::GBTreeModel& model, size_t tree_begin, size_t tree_end) {
    CHECK_EQ(model.param.size_leaf_vector, 0);
    // Copy decision trees to device
    thrust::host_vector<size_t> h_tree_segments{};
    h_tree_segments.reserve((tree_end - tree_begin) + 1);
    size_t sum = 0;
    h_tree_segments.push_back(sum);
    for (auto tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      sum += model.trees.at(tree_idx)->GetNodes().size();
      h_tree_segments.push_back(sum);
    }

    thrust::host_vector<DevicePredictionNode> h_nodes(h_tree_segments.back());
    for (auto tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      auto& src_nodes = model.trees.at(tree_idx)->GetNodes();
      std::copy(src_nodes.begin(), src_nodes.end(),
                h_nodes.begin() + h_tree_segments[tree_idx - tree_begin]);
    }
    InitModel(model, h_tree_segments, h_nodes, tree_begin, tree_end);
  }

  void DevicePredictInternal(DMatrix* dmat,
                             HostDeviceVector<bst_float>* out_preds,
                             const gbm::GBTreeModel& model, size_t tree_begin,
                             size_t tree_end) {
    if (tree_end - tree_begin == 0) {
      return;
    }
    monitor_.StartCuda("DevicePredictInternal");

    InitModel(model, tree_begin, tree_end);

    size_t batch_offset = 0;
    for (auto &batch : dmat->GetBatches<SparsePage>()) {
      batch.offset.SetDevice(generic_param_->gpu_id);
      batch.data.SetDevice(generic_param_->gpu_id);
      PredictInternal(batch, model.learner_model_param_->num_feature,
                      out_preds, batch_offset);
      batch_offset += batch.Size() * model.learner_model_param_->num_output_group;
    }

    monitor_.StopCuda("DevicePredictInternal");
  }

 public:
  GPUPredictor(GenericParameter const* generic_param,
               std::shared_ptr<std::unordered_map<DMatrix*, PredictionCacheEntry>> cache) :
      Predictor::Predictor{generic_param, cache} {}

  ~GPUPredictor() override {
    if (generic_param_->gpu_id >= 0) {
      dh::safe_cuda(hipSetDevice(generic_param_->gpu_id));
    }
  }

  void PredictBatch(DMatrix* dmat, HostDeviceVector<bst_float>* out_preds,
                    const gbm::GBTreeModel& model, int tree_begin,
                    unsigned ntree_limit = 0) override {
    int device = generic_param_->gpu_id;
    CHECK_GE(device, 0) << "Set `gpu_id' to positive value for processing GPU data.";
    ConfigureDevice(device);

    if (this->PredictFromCache(dmat, out_preds, model, ntree_limit)) {
      return;
    }
    this->InitOutPredictions(dmat->Info(), out_preds, model);

    int32_t tree_end = ntree_limit * model.learner_model_param_->num_output_group;

    if (ntree_limit == 0 || ntree_limit > model.trees.size()) {
      tree_end = static_cast<unsigned>(model.trees.size());
    }

    DevicePredictInternal(dmat, out_preds, model, tree_begin, tree_end);

    auto cache_emtry = this->FindCache(dmat);
    if (cache_emtry == cache_->cend()) { return; }
    if (cache_emtry->second.predictions.Size() == 0) {
      // Initialise the cache on first iteration, this comes useful
      // when performing training continuation:
      //
      // 1. PredictBatch
      // 2. CommitModel
      //  - updater->UpdatePredictionCache
      //
      // If we don't initialise this cache, the 2 step will recieve an invalid cache as
      // the first step only modifies prediction store in learner without following code.
      InitOutPredictions(cache_emtry->second.data->Info(),
                         &(cache_emtry->second.predictions), model);
      CHECK_EQ(cache_emtry->second.predictions.Size(), out_preds->Size());
      cache_emtry->second.predictions.Copy(*out_preds);
    }
  }

 protected:
  void InitOutPredictions(const MetaInfo& info,
                          HostDeviceVector<bst_float>* out_preds,
                          const gbm::GBTreeModel& model) const {
    size_t n_classes = model.learner_model_param_->num_output_group;
    size_t n = n_classes * info.num_row_;
    const HostDeviceVector<bst_float>& base_margin = info.base_margin_;
    out_preds->SetDevice(generic_param_->gpu_id);
    out_preds->Resize(n);
    if (base_margin.Size() != 0) {
      CHECK_EQ(base_margin.Size(), n);
      out_preds->Copy(base_margin);
    } else {
      out_preds->Fill(model.learner_model_param_->base_score);
    }
  }

  bool PredictFromCache(DMatrix* dmat, HostDeviceVector<bst_float>* out_preds,
                        const gbm::GBTreeModel& model, unsigned ntree_limit) {
    if (ntree_limit == 0 ||
        ntree_limit * model.learner_model_param_->num_output_group >= model.trees.size()) {
      auto it = (*cache_).find(dmat);
      if (it != cache_->cend()) {
        const HostDeviceVector<bst_float>& y = it->second.predictions;
        if (y.Size() != 0) {
          monitor_.StartCuda("PredictFromCache");
          out_preds->SetDevice(y.DeviceIdx());
          out_preds->Resize(y.Size());
          out_preds->Copy(y);
          monitor_.StopCuda("PredictFromCache");
          return true;
        }
      }
    }
    return false;
  }

  void UpdatePredictionCache(
      const gbm::GBTreeModel& model,
      std::vector<std::unique_ptr<TreeUpdater>>* updaters,
      int num_new_trees) override {
    auto old_ntree = model.trees.size() - num_new_trees;
    // update cache entry
    for (auto& kv : (*cache_)) {
      PredictionCacheEntry& e = kv.second;
      DMatrix* dmat = kv.first;
      HostDeviceVector<bst_float>& predictions = e.predictions;

      if (predictions.Size() == 0) {
        this->InitOutPredictions(dmat->Info(), &predictions, model);
      }

      if (model.learner_model_param_->num_output_group == 1 && updaters->size() > 0 &&
          num_new_trees == 1 &&
          updaters->back()->UpdatePredictionCache(e.data.get(), &predictions)) {
        // do nothing
      } else {
        DevicePredictInternal(dmat, &predictions, model, old_ntree, model.trees.size());
      }
    }
  }

  void PredictInstance(const SparsePage::Inst& inst,
                       std::vector<bst_float>* out_preds,
                       const gbm::GBTreeModel& model, unsigned ntree_limit) override {
    LOG(FATAL) << "[Internal error]: " << __func__
               << " is not implemented in GPU Predictor.";
  }

  void PredictLeaf(DMatrix* p_fmat, std::vector<bst_float>* out_preds,
                   const gbm::GBTreeModel& model,
                   unsigned ntree_limit) override {
    LOG(FATAL) << "[Internal error]: " << __func__
               << " is not implemented in GPU Predictor.";
  }

  void PredictContribution(DMatrix* p_fmat,
                           std::vector<bst_float>* out_contribs,
                           const gbm::GBTreeModel& model, unsigned ntree_limit,
                           std::vector<bst_float>* tree_weights,
                           bool approximate, int condition,
                           unsigned condition_feature) override {
    LOG(FATAL) << "[Internal error]: " << __func__
               << " is not implemented in GPU Predictor.";
  }

  void PredictInteractionContributions(DMatrix* p_fmat,
                                       std::vector<bst_float>* out_contribs,
                                       const gbm::GBTreeModel& model,
                                       unsigned ntree_limit,
                                       std::vector<bst_float>* tree_weights,
                                       bool approximate) override {
    LOG(FATAL) << "[Internal error]: " << __func__
               << " is not implemented in GPU Predictor.";
  }

  void Configure(const std::vector<std::pair<std::string, std::string>>& cfg) override {
    Predictor::Configure(cfg);

    int device = generic_param_->gpu_id;
    if (device >= 0) {
      ConfigureDevice(device);
    }
  }

 private:
  /*! \brief Reconfigure the device when GPU is changed. */
  void ConfigureDevice(int device) {
    if (device >= 0) {
      max_shared_memory_bytes_ = dh::MaxSharedMemory(device);
    }
  }

  common::Monitor monitor_;
  dh::device_vector<DevicePredictionNode> nodes_;
  dh::device_vector<size_t> tree_segments_;
  dh::device_vector<int> tree_group_;
  size_t max_shared_memory_bytes_;
  size_t tree_begin_;
  size_t tree_end_;
  int num_group_;
};

XGBOOST_REGISTER_PREDICTOR(GPUPredictor, "gpu_predictor")
.describe("Make predictions using GPU.")
.set_body([](GenericParameter const* generic_param,
             std::shared_ptr<std::unordered_map<DMatrix*, PredictionCacheEntry>> cache) {
            return new GPUPredictor(generic_param, cache);
          });

}  // namespace predictor
}  // namespace xgboost
