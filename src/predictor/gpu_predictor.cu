#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2021 by Contributors
 */
#include <GPUTreeShap/gpu_treeshap.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>

#include <memory>

#include "../common/bitfield.h"
#include "../common/categorical.h"
#include "../common/common.h"
#include "../common/device_helpers.cuh"
#include "../data/device_adapter.cuh"
#include "../data/ellpack_page.cuh"
#include "../data/proxy_dmatrix.h"
#include "../gbm/gbtree_model.h"
#include "predict_fn.h"
#include "xgboost/data.h"
#include "xgboost/host_device_vector.h"
#include "xgboost/predictor.h"
#include "xgboost/tree_model.h"
#include "xgboost/tree_updater.h"

namespace xgboost {
namespace predictor {

DMLC_REGISTRY_FILE_TAG(gpu_predictor);

struct TreeView {
  RegTree::CategoricalSplitMatrix cats;
  common::Span<RegTree::Node const> d_tree;

  XGBOOST_DEVICE
  TreeView(size_t tree_begin, size_t tree_idx,
           common::Span<const RegTree::Node> d_nodes,
           common::Span<size_t const> d_tree_segments,
           common::Span<FeatureType const> d_tree_split_types,
           common::Span<uint32_t const> d_cat_tree_segments,
           common::Span<RegTree::Segment const> d_cat_node_segments,
           common::Span<uint32_t const> d_categories) {
    auto begin = d_tree_segments[tree_idx - tree_begin];
    auto n_nodes = d_tree_segments[tree_idx - tree_begin + 1] -
                   d_tree_segments[tree_idx - tree_begin];

    d_tree = d_nodes.subspan(begin, n_nodes);

    auto tree_cat_ptrs = d_cat_node_segments.subspan(begin, n_nodes);
    auto tree_split_types = d_tree_split_types.subspan(begin, n_nodes);

    auto tree_categories =
        d_categories.subspan(d_cat_tree_segments[tree_idx - tree_begin],
                             d_cat_tree_segments[tree_idx - tree_begin + 1] -
                                 d_cat_tree_segments[tree_idx - tree_begin]);

    cats.split_type = tree_split_types;
    cats.categories = tree_categories;
    cats.node_ptr = tree_cat_ptrs;
  }

  __device__ bool HasCategoricalSplit() const {
    return !cats.categories.empty();
  }
};

struct SparsePageView {
  common::Span<const Entry> d_data;
  common::Span<const bst_row_t> d_row_ptr;
  bst_feature_t num_features;

  SparsePageView() = default;
  XGBOOST_DEVICE SparsePageView(common::Span<const Entry> data,
                                common::Span<const bst_row_t> row_ptr,
                                bst_feature_t num_features)
      : d_data{data}, d_row_ptr{row_ptr}, num_features(num_features) {}
  __device__ float GetElement(size_t ridx, size_t fidx) const {
    // Binary search
    auto begin_ptr = d_data.begin() + d_row_ptr[ridx];
    auto end_ptr = d_data.begin() + d_row_ptr[ridx + 1];
    if (end_ptr - begin_ptr == this->NumCols()) {
      // Bypass span check for dense data
      return d_data.data()[d_row_ptr[ridx] + fidx].fvalue;
    }
    common::Span<const Entry>::iterator previous_middle;
    while (end_ptr != begin_ptr) {
      auto middle = begin_ptr + (end_ptr - begin_ptr) / 2;
      if (middle == previous_middle) {
        break;
      } else {
        previous_middle = middle;
      }

      if (middle->index == fidx) {
        return middle->fvalue;
      } else if (middle->index < fidx) {
        begin_ptr = middle;
      } else {
        end_ptr = middle;
      }
    }
    // Value is missing
    return nanf("");
  }
  XGBOOST_DEVICE size_t NumRows() const { return d_row_ptr.size() - 1; }
  XGBOOST_DEVICE size_t NumCols() const { return num_features; }
};

struct SparsePageLoader {
  bool use_shared;
  SparsePageView data;
  float* smem;
  size_t entry_start;

  __device__ SparsePageLoader(SparsePageView data, bool use_shared, bst_feature_t num_features,
                              bst_row_t num_rows, size_t entry_start, float)
      : use_shared(use_shared),
        data(data),
        entry_start(entry_start) {
    extern __shared__ float _smem[];
    smem = _smem;
    // Copy instances
    if (use_shared) {
      bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
      int shared_elements = blockDim.x * data.num_features;
      dh::BlockFill(smem, shared_elements, nanf(""));
      __syncthreads();
      if (global_idx < num_rows) {
        bst_uint elem_begin = data.d_row_ptr[global_idx];
        bst_uint elem_end = data.d_row_ptr[global_idx + 1];
        for (bst_uint elem_idx = elem_begin; elem_idx < elem_end; elem_idx++) {
          Entry elem = data.d_data[elem_idx - entry_start];
          smem[threadIdx.x * data.num_features + elem.index] = elem.fvalue;
        }
      }
      __syncthreads();
    }
  }
  __device__ float GetElement(size_t  ridx, size_t  fidx) const {
    if (use_shared) {
      return smem[threadIdx.x * data.num_features + fidx];
    } else {
      return data.GetElement(ridx, fidx);
    }
  }
};

struct EllpackLoader {
  EllpackDeviceAccessor const& matrix;
  XGBOOST_DEVICE EllpackLoader(EllpackDeviceAccessor const& m, bool, bst_feature_t, bst_row_t,
                               size_t, float)
      : matrix{m} {}
  __device__ __forceinline__ float GetElement(size_t ridx, size_t fidx) const {
    auto gidx = matrix.GetBinIndex(ridx, fidx);
    if (gidx == -1) {
      return nan("");
    }
    if (common::IsCat(matrix.feature_types, fidx)) {
      return matrix.gidx_fvalue_map[gidx];
    }
    // The gradient index needs to be shifted by one as min values are not included in the
    // cuts.
    if (gidx == matrix.feature_segments[fidx]) {
      return matrix.min_fvalue[fidx];
    }
    return matrix.gidx_fvalue_map[gidx - 1];
  }
};

template <typename Batch>
struct DeviceAdapterLoader {
  Batch batch;
  bst_feature_t columns;
  float* smem;
  bool use_shared;
  data::IsValidFunctor is_valid;

  using BatchT = Batch;

  XGBOOST_DEV_INLINE DeviceAdapterLoader(Batch const batch, bool use_shared,
                                         bst_feature_t num_features, bst_row_t num_rows,
                                         size_t entry_start, float missing) :
    batch{batch},
    columns{num_features},
    use_shared{use_shared},
    is_valid{missing} {
      extern __shared__ float _smem[];
      smem = _smem;
      if (use_shared) {
        uint32_t global_idx = blockDim.x * blockIdx.x + threadIdx.x;
        size_t shared_elements = blockDim.x * num_features;
        dh::BlockFill(smem, shared_elements, nanf(""));
        __syncthreads();
        if (global_idx < num_rows) {
          auto beg = global_idx * columns;
          auto end = (global_idx + 1) * columns;
          for (size_t i = beg; i < end; ++i) {
            auto value = batch.GetElement(i).value;
            if (is_valid(value)) {
              smem[threadIdx.x * num_features + (i - beg)] = value;
            }
          }
        }
      }
      __syncthreads();
    }

  XGBOOST_DEV_INLINE  float GetElement(size_t  ridx, size_t  fidx) const {
    if (use_shared) {
      return smem[threadIdx.x * columns + fidx];
    }
    auto value = batch.GetElement(ridx * columns + fidx).value;
    if (is_valid(value)) {
      return value;
    } else {
      return nan("");
    }
  }
};

template <bool has_missing, bool has_categorical, typename Loader>
__device__ bst_node_t GetLeafIndex(bst_row_t ridx, TreeView const &tree,
                                   Loader *loader) {
  bst_node_t nidx = 0;
  RegTree::Node n = tree.d_tree[nidx];
  while (!n.IsLeaf()) {
    float fvalue = loader->GetElement(ridx, n.SplitIndex());
    bool is_missing = common::CheckNAN(fvalue);
    nidx = GetNextNode<has_missing, has_categorical>(n, nidx, fvalue,
                                                     is_missing, tree.cats);
    n = tree.d_tree[nidx];
  }
  return nidx;
}

template <bool has_missing, typename Loader>
__device__ float GetLeafWeight(bst_row_t ridx, TreeView const &tree,
                               Loader *loader) {
  bst_node_t nidx = -1;
  if (tree.HasCategoricalSplit()) {
    nidx = GetLeafIndex<has_missing, true>(ridx, tree, loader);
  } else {
    nidx = GetLeafIndex<has_missing, false>(ridx, tree, loader);
  }
  return tree.d_tree[nidx].LeafValue();
}

template <typename Loader, typename Data>
__global__ void
PredictLeafKernel(Data data, common::Span<const RegTree::Node> d_nodes,
                  common::Span<float> d_out_predictions,
                  common::Span<size_t const> d_tree_segments,

                  common::Span<FeatureType const> d_tree_split_types,
                  common::Span<uint32_t const> d_cat_tree_segments,
                  common::Span<RegTree::Segment const> d_cat_node_segments,
                  common::Span<uint32_t const> d_categories,

                  size_t tree_begin, size_t tree_end, size_t num_features,
                  size_t num_rows, size_t entry_start, bool use_shared,
                  float missing) {
  bst_row_t ridx = blockDim.x * blockIdx.x + threadIdx.x;
  if (ridx >= num_rows) {
    return;
  }
  Loader loader(data, use_shared, num_features, num_rows, entry_start, missing);
  for (size_t tree_idx = tree_begin; tree_idx < tree_end; ++tree_idx) {
    TreeView d_tree{
        tree_begin,          tree_idx,           d_nodes,
        d_tree_segments,     d_tree_split_types, d_cat_tree_segments,
        d_cat_node_segments, d_categories};

    bst_node_t leaf = -1;
    if (d_tree.HasCategoricalSplit()) {
      leaf = GetLeafIndex<true, true>(ridx, d_tree, &loader);
    } else {
      leaf = GetLeafIndex<true, false>(ridx, d_tree, &loader);
    }
    d_out_predictions[ridx * (tree_end - tree_begin) + tree_idx] = leaf;
  }
}

template <typename Loader, typename Data, bool has_missing = true>
__global__ void
PredictKernel(Data data, common::Span<const RegTree::Node> d_nodes,
              common::Span<float> d_out_predictions,
              common::Span<size_t const> d_tree_segments,
              common::Span<int const> d_tree_group,
              common::Span<FeatureType const> d_tree_split_types,
              common::Span<uint32_t const> d_cat_tree_segments,
              common::Span<RegTree::Segment const> d_cat_node_segments,
              common::Span<uint32_t const> d_categories, size_t tree_begin,
              size_t tree_end, size_t num_features, size_t num_rows,
              size_t entry_start, bool use_shared, int num_group, float missing) {
  bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
  Loader loader(data, use_shared, num_features, num_rows, entry_start, missing);
  if (global_idx >= num_rows) return;
  if (num_group == 1) {
    float sum = 0;
    for (size_t tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      TreeView d_tree{
          tree_begin,          tree_idx,           d_nodes,
          d_tree_segments,     d_tree_split_types, d_cat_tree_segments,
          d_cat_node_segments, d_categories};
      float leaf = GetLeafWeight<has_missing>(global_idx, d_tree, &loader);
      sum += leaf;
    }
    d_out_predictions[global_idx] += sum;
  } else {
    for (size_t tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      int tree_group = d_tree_group[tree_idx];
      TreeView d_tree{
          tree_begin,          tree_idx,           d_nodes,
          d_tree_segments,     d_tree_split_types, d_cat_tree_segments,
          d_cat_node_segments, d_categories};
      bst_uint out_prediction_idx = global_idx * num_group + tree_group;
      d_out_predictions[out_prediction_idx] +=
          GetLeafWeight<has_missing>(global_idx, d_tree, &loader);
    }
  }
}

class DeviceModel {
 public:
  // Need to lazily construct the vectors because GPU id is only known at runtime
  HostDeviceVector<RTreeNodeStat> stats;
  HostDeviceVector<size_t> tree_segments;
  HostDeviceVector<RegTree::Node> nodes;
  HostDeviceVector<int> tree_group;
  HostDeviceVector<FeatureType> split_types;

  // Pointer to each tree, segmenting the node array.
  HostDeviceVector<uint32_t> categories_tree_segments;
  // Pointer to each node, segmenting categories array.
  HostDeviceVector<RegTree::Segment> categories_node_segments;
  HostDeviceVector<uint32_t> categories;

  size_t tree_beg_;  // NOLINT
  size_t tree_end_;  // NOLINT
  int num_group;

  void Init(const gbm::GBTreeModel& model, size_t tree_begin, size_t tree_end, int32_t gpu_id) {
    dh::safe_cuda(hipSetDevice(gpu_id));

    CHECK_EQ(model.param.size_leaf_vector, 0);
    // Copy decision trees to device
    tree_segments = std::move(HostDeviceVector<size_t>({}, gpu_id));
    auto& h_tree_segments = tree_segments.HostVector();
    h_tree_segments.reserve((tree_end - tree_begin) + 1);
    size_t sum = 0;
    h_tree_segments.push_back(sum);
    for (auto tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      sum += model.trees.at(tree_idx)->GetNodes().size();
      h_tree_segments.push_back(sum);
    }

    nodes = std::move(HostDeviceVector<RegTree::Node>(h_tree_segments.back(), RegTree::Node(),
                                                      gpu_id));
    stats = std::move(HostDeviceVector<RTreeNodeStat>(h_tree_segments.back(),
                                                      RTreeNodeStat(), gpu_id));
    auto d_nodes = nodes.DevicePointer();
    auto d_stats = stats.DevicePointer();
    for (auto tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      auto& src_nodes = model.trees.at(tree_idx)->GetNodes();
      auto& src_stats = model.trees.at(tree_idx)->GetStats();
      dh::safe_cuda(hipMemcpyAsync(
          d_nodes + h_tree_segments[tree_idx - tree_begin], src_nodes.data(),
          sizeof(RegTree::Node) * src_nodes.size(), hipMemcpyDefault));
      dh::safe_cuda(hipMemcpyAsync(
          d_stats + h_tree_segments[tree_idx - tree_begin], src_stats.data(),
          sizeof(RTreeNodeStat) * src_stats.size(), hipMemcpyDefault));
    }

    tree_group = std::move(HostDeviceVector<int>(model.tree_info.size(), 0, gpu_id));
    auto& h_tree_group = tree_group.HostVector();
    std::memcpy(h_tree_group.data(), model.tree_info.data(), sizeof(int) * model.tree_info.size());

    // Initialize categorical splits.
    split_types.SetDevice(gpu_id);
    std::vector<FeatureType>& h_split_types = split_types.HostVector();
    h_split_types.resize(h_tree_segments.back());
    for (auto tree_idx = tree_begin; tree_idx < tree_end; ++tree_idx) {
      auto const& src_st = model.trees.at(tree_idx)->GetSplitTypes();
      std::copy(src_st.cbegin(), src_st.cend(),
                h_split_types.begin() + h_tree_segments[tree_idx - tree_begin]);
    }

    categories = HostDeviceVector<uint32_t>({}, gpu_id);
    categories_tree_segments = HostDeviceVector<uint32_t>(1, 0, gpu_id);
    std::vector<uint32_t> &h_categories = categories.HostVector();
    std::vector<uint32_t> &h_split_cat_segments = categories_tree_segments.HostVector();
    for (auto tree_idx = tree_begin; tree_idx < tree_end; ++tree_idx) {
      auto const& src_cats = model.trees.at(tree_idx)->GetSplitCategories();
      size_t orig_size = h_categories.size();
      h_categories.resize(orig_size + src_cats.size());
      std::copy(src_cats.cbegin(), src_cats.cend(),
                h_categories.begin() + orig_size);
      h_split_cat_segments.push_back(h_categories.size());
    }

    categories_node_segments =
        HostDeviceVector<RegTree::Segment>(h_tree_segments.back(), {}, gpu_id);
    std::vector<RegTree::Segment> &h_categories_node_segments =
        categories_node_segments.HostVector();
    for (auto tree_idx = tree_begin; tree_idx < tree_end; ++tree_idx) {
      auto const &src_cats_ptr = model.trees.at(tree_idx)->GetSplitCategoriesPtr();
      std::copy(src_cats_ptr.cbegin(), src_cats_ptr.cend(),
                h_categories_node_segments.begin() +
                    h_tree_segments[tree_idx - tree_begin]);
    }

    this->tree_beg_ = tree_begin;
    this->tree_end_ = tree_end;
    this->num_group = model.learner_model_param->num_output_group;
  }
};

struct ShapSplitCondition {
  ShapSplitCondition() = default;
  XGBOOST_DEVICE
  ShapSplitCondition(float feature_lower_bound, float feature_upper_bound,
                     bool is_missing_branch, common::CatBitField cats)
      : feature_lower_bound(feature_lower_bound),
        feature_upper_bound(feature_upper_bound),
        is_missing_branch(is_missing_branch), categories{std::move(cats)} {
    assert(feature_lower_bound <= feature_upper_bound);
  }

  /*! Feature values >= lower and < upper flow down this path. */
  float feature_lower_bound;
  float feature_upper_bound;
  /*! Feature value set to true flow down this path. */
  common::CatBitField categories;
  /*! Do missing values flow down this path? */
  bool is_missing_branch;

  // Does this instance flow down this path?
  XGBOOST_DEVICE bool EvaluateSplit(float x) const {
    // is nan
    if (isnan(x)) {
      return is_missing_branch;
    }
    if (categories.Size() != 0) {
      auto cat = static_cast<uint32_t>(x);
      return categories.Check(cat);
    } else {
      return x >= feature_lower_bound && x < feature_upper_bound;
    }
  }

  // the &= op in bitfiled is per cuda thread, this one loops over the entire
  // bitfield.
  XGBOOST_DEVICE static common::CatBitField Intersect(common::CatBitField l,
                                                      common::CatBitField r) {
    if (l.Data() == r.Data()) {
      return l;
    }
    if (l.Size() > r.Size()) {
      thrust::swap(l, r);
    }
    for (size_t i = 0; i < r.Bits().size(); ++i) {
      l.Bits()[i] &= r.Bits()[i];
    }
    return l;
  }

  // Combine two split conditions on the same feature
  XGBOOST_DEVICE void Merge(ShapSplitCondition other) {
    // Combine duplicate features
    if (categories.Size() != 0 || other.categories.Size() != 0) {
      categories = Intersect(categories, other.categories);
    } else {
      feature_lower_bound = max(feature_lower_bound, other.feature_lower_bound);
      feature_upper_bound = min(feature_upper_bound, other.feature_upper_bound);
    }
    is_missing_branch = is_missing_branch && other.is_missing_branch;
  }
};

struct PathInfo {
  int64_t leaf_position;  // -1 not a leaf
  size_t length;
  size_t tree_idx;
};

// Transform model into path element form for GPUTreeShap
void ExtractPaths(
    dh::device_vector<gpu_treeshap::PathElement<ShapSplitCondition>> *paths,
    DeviceModel *model, dh::device_vector<uint32_t> *path_categories,
    int gpu_id) {
  dh::safe_cuda(hipSetDevice(gpu_id));
  auto& device_model = *model;

  dh::caching_device_vector<PathInfo> info(device_model.nodes.Size());
  dh::XGBCachingDeviceAllocator<PathInfo> alloc;
  auto d_nodes = device_model.nodes.ConstDeviceSpan();
  auto d_tree_segments = device_model.tree_segments.ConstDeviceSpan();
  auto nodes_transform = dh::MakeTransformIterator<PathInfo>(
      thrust::make_counting_iterator(0ull), [=] __device__(size_t idx) {
        auto n = d_nodes[idx];
        if (!n.IsLeaf() || n.IsDeleted()) {
          return PathInfo{-1, 0, 0};
        }
        size_t tree_idx =
            dh::SegmentId(d_tree_segments.begin(), d_tree_segments.end(), idx);
        size_t tree_offset = d_tree_segments[tree_idx];
        size_t path_length = 1;
        while (!n.IsRoot()) {
          n = d_nodes[n.Parent() + tree_offset];
          path_length++;
        }
        return PathInfo{int64_t(idx), path_length, tree_idx};
      });
  auto end = thrust::copy_if(
      thrust::cuda::par(alloc), nodes_transform,
      nodes_transform + d_nodes.size(), info.begin(),
      [=] __device__(const PathInfo& e) { return e.leaf_position != -1; });
  info.resize(end - info.begin());
  auto length_iterator = dh::MakeTransformIterator<size_t>(
      info.begin(),
      [=] __device__(const PathInfo& info) { return info.length; });
  dh::caching_device_vector<size_t> path_segments(info.size() + 1);
  thrust::exclusive_scan(thrust::cuda::par(alloc), length_iterator,
                         length_iterator + info.size() + 1,
                         path_segments.begin());

  paths->resize(path_segments.back());

  auto d_paths = dh::ToSpan(*paths);
  auto d_info = info.data().get();
  auto d_stats = device_model.stats.ConstDeviceSpan();
  auto d_tree_group = device_model.tree_group.ConstDeviceSpan();
  auto d_path_segments = path_segments.data().get();

  auto d_split_types = device_model.split_types.ConstDeviceSpan();
  auto d_cat_segments = device_model.categories_tree_segments.ConstDeviceSpan();
  auto d_cat_node_segments = device_model.categories_node_segments.ConstDeviceSpan();

  size_t max_cat = 0;
  if (thrust::any_of(dh::tbegin(d_split_types), dh::tend(d_split_types),
                     common::IsCatOp{})) {
    dh::PinnedMemory pinned;
    auto h_max_cat = pinned.GetSpan<RegTree::Segment>(1);
    auto max_elem_it = dh::MakeTransformIterator<size_t>(
        dh::tbegin(d_cat_node_segments),
        [] __device__(RegTree::Segment seg) { return seg.size; });
    size_t max_cat_it =
        thrust::max_element(thrust::device, max_elem_it,
                            max_elem_it + d_cat_node_segments.size()) -
        max_elem_it;
    dh::safe_cuda(hipMemcpy(h_max_cat.data(),
                             d_cat_node_segments.data() + max_cat_it,
                             h_max_cat.size_bytes(), hipMemcpyDeviceToHost));
    max_cat = h_max_cat[0].size;
    CHECK_GE(max_cat, 1);
    path_categories->resize(max_cat * paths->size());
  }

  auto d_model_categories = device_model.categories.DeviceSpan();
  common::Span<uint32_t> d_path_categories = dh::ToSpan(*path_categories);

  dh::LaunchN(info.size(), [=] __device__(size_t idx) {
    auto path_info = d_info[idx];
    size_t tree_offset = d_tree_segments[path_info.tree_idx];
    TreeView tree{0,                   path_info.tree_idx, d_nodes,
                  d_tree_segments,     d_split_types,      d_cat_segments,
                  d_cat_node_segments, d_model_categories};
    int group = d_tree_group[path_info.tree_idx];
    size_t child_idx = path_info.leaf_position;
    auto child = d_nodes[child_idx];
    float v = child.LeafValue();
    const float inf = std::numeric_limits<float>::infinity();
    size_t output_position = d_path_segments[idx + 1] - 1;
    while (!child.IsRoot()) {
      size_t parent_idx = tree_offset + child.Parent();
      double child_cover = d_stats[child_idx].sum_hess;
      double parent_cover = d_stats[parent_idx].sum_hess;
      double zero_fraction = child_cover / parent_cover;
      auto parent = tree.d_tree[child.Parent()];

      bool is_left_path = (tree_offset + parent.LeftChild()) == child_idx;
      bool is_missing_path = (!parent.DefaultLeft() && !is_left_path) ||
                             (parent.DefaultLeft() && is_left_path);

      float lower_bound = -inf;
      float upper_bound = inf;
      common::CatBitField bits;
      if (common::IsCat(tree.cats.split_type, child.Parent())) {
        auto path_cats = d_path_categories.subspan(max_cat * output_position, max_cat);
        size_t size = tree.cats.node_ptr[child.Parent()].size;
        auto node_cats = tree.cats.categories.subspan(tree.cats.node_ptr[child.Parent()].beg, size);
        SPAN_CHECK(path_cats.size() >= node_cats.size());
        for (size_t i = 0; i < node_cats.size(); ++i) {
          path_cats[i] = is_left_path ? ~node_cats[i] : node_cats[i];
        }
        bits = common::CatBitField{path_cats};
      } else {
        lower_bound = is_left_path ? -inf : parent.SplitCond();
        upper_bound = is_left_path ? parent.SplitCond() : inf;
      }
      d_paths[output_position--] =
          gpu_treeshap::PathElement<ShapSplitCondition>{
              idx,           parent.SplitIndex(),
              group,         ShapSplitCondition{lower_bound, upper_bound, is_missing_path, bits},
              zero_fraction, v};
      child_idx = parent_idx;
      child = parent;
    }
    // Root node has feature -1
    d_paths[output_position] = {idx, -1, group, ShapSplitCondition{-inf, inf, false, {}}, 1.0, v};
  });
}

namespace {
template <size_t kBlockThreads>
size_t SharedMemoryBytes(size_t cols, size_t max_shared_memory_bytes) {
  // No way max_shared_memory_bytes that is equal to 0.
  CHECK_GT(max_shared_memory_bytes, 0);
  size_t shared_memory_bytes =
      static_cast<size_t>(sizeof(float) * cols * kBlockThreads);
  if (shared_memory_bytes > max_shared_memory_bytes) {
    shared_memory_bytes = 0;
  }
  return shared_memory_bytes;
}
}  // anonymous namespace

class GPUPredictor : public xgboost::Predictor {
 private:
  void PredictInternal(const SparsePage& batch,
                       DeviceModel const& model,
                       size_t num_features,
                       HostDeviceVector<bst_float>* predictions,
                       size_t batch_offset, bool is_dense) const {
    batch.offset.SetDevice(ctx_->gpu_id);
    batch.data.SetDevice(ctx_->gpu_id);
    const uint32_t BLOCK_THREADS = 128;
    size_t num_rows = batch.Size();
    auto GRID_SIZE = static_cast<uint32_t>(common::DivRoundUp(num_rows, BLOCK_THREADS));
    auto max_shared_memory_bytes = ConfigureDevice(ctx_->gpu_id);
    size_t shared_memory_bytes =
        SharedMemoryBytes<BLOCK_THREADS>(num_features, max_shared_memory_bytes);
    bool use_shared = shared_memory_bytes != 0;

    size_t entry_start = 0;
    SparsePageView data(batch.data.DeviceSpan(), batch.offset.DeviceSpan(),
                        num_features);
    auto const kernel = [&](auto predict_fn) {
      dh::LaunchKernel {GRID_SIZE, BLOCK_THREADS, shared_memory_bytes} (
          predict_fn, data, model.nodes.ConstDeviceSpan(),
          predictions->DeviceSpan().subspan(batch_offset),
          model.tree_segments.ConstDeviceSpan(),
          model.tree_group.ConstDeviceSpan(),
          model.split_types.ConstDeviceSpan(),
          model.categories_tree_segments.ConstDeviceSpan(),
          model.categories_node_segments.ConstDeviceSpan(),
          model.categories.ConstDeviceSpan(), model.tree_beg_, model.tree_end_,
          num_features, num_rows, entry_start, use_shared, model.num_group,
          nan(""));
    };
    if (is_dense) {
      kernel(PredictKernel<SparsePageLoader, SparsePageView, false>);
    } else {
      kernel(PredictKernel<SparsePageLoader, SparsePageView, true>);
    }
  }
  void PredictInternal(EllpackDeviceAccessor const& batch,
                       DeviceModel const& model,
                       HostDeviceVector<bst_float>* out_preds,
                       size_t batch_offset) const {
    const uint32_t BLOCK_THREADS = 256;
    size_t num_rows = batch.n_rows;
    auto GRID_SIZE = static_cast<uint32_t>(common::DivRoundUp(num_rows, BLOCK_THREADS));
    DeviceModel d_model;

    bool use_shared = false;
    size_t entry_start = 0;
    dh::LaunchKernel {GRID_SIZE, BLOCK_THREADS} (
        PredictKernel<EllpackLoader, EllpackDeviceAccessor>, batch,
        model.nodes.ConstDeviceSpan(), out_preds->DeviceSpan().subspan(batch_offset),
        model.tree_segments.ConstDeviceSpan(), model.tree_group.ConstDeviceSpan(),
        model.split_types.ConstDeviceSpan(),
        model.categories_tree_segments.ConstDeviceSpan(),
        model.categories_node_segments.ConstDeviceSpan(),
        model.categories.ConstDeviceSpan(), model.tree_beg_, model.tree_end_,
        batch.NumFeatures(), num_rows, entry_start, use_shared,
        model.num_group, nan(""));
  }

  void DevicePredictInternal(DMatrix* dmat, HostDeviceVector<float>* out_preds,
                             const gbm::GBTreeModel& model, size_t tree_begin,
                             size_t tree_end) const {
    if (tree_end - tree_begin == 0) {
      return;
    }
    out_preds->SetDevice(ctx_->gpu_id);
    auto const& info = dmat->Info();
    DeviceModel d_model;
    d_model.Init(model, tree_begin, tree_end, ctx_->gpu_id);

    if (dmat->PageExists<SparsePage>()) {
      size_t batch_offset = 0;
      for (auto &batch : dmat->GetBatches<SparsePage>()) {
        this->PredictInternal(batch, d_model, model.learner_model_param->num_feature,
                              out_preds, batch_offset, dmat->IsDense());
        batch_offset += batch.Size() * model.learner_model_param->num_output_group;
      }
    } else {
      size_t batch_offset = 0;
      for (auto const& page : dmat->GetBatches<EllpackPage>(BatchParam{})) {
        dmat->Info().feature_types.SetDevice(ctx_->gpu_id);
        auto feature_types = dmat->Info().feature_types.ConstDeviceSpan();
        this->PredictInternal(
            page.Impl()->GetDeviceAccessor(ctx_->gpu_id, feature_types),
            d_model,
            out_preds,
            batch_offset);
        batch_offset += page.Impl()->n_rows;
      }
    }
  }

 public:
  explicit GPUPredictor(GenericParameter const* generic_param) :
      Predictor::Predictor{generic_param} {}

  ~GPUPredictor() override {
    if (ctx_->gpu_id >= 0 && ctx_->gpu_id < common::AllVisibleGPUs()) {
      dh::safe_cuda(hipSetDevice(ctx_->gpu_id));
    }
  }

  void PredictBatch(DMatrix* dmat, PredictionCacheEntry* predts,
                    const gbm::GBTreeModel& model, uint32_t tree_begin,
                    uint32_t tree_end = 0) const override {
    int device = ctx_->gpu_id;
    CHECK_GE(device, 0) << "Set `gpu_id' to positive value for processing GPU data.";
    auto* out_preds = &predts->predictions;
    if (tree_end == 0) {
      tree_end = model.trees.size();
    }
    this->DevicePredictInternal(dmat, out_preds, model, tree_begin, tree_end);
  }

  template <typename Adapter, typename Loader>
  void DispatchedInplacePredict(dmlc::any const &x, std::shared_ptr<DMatrix> p_m,
                                const gbm::GBTreeModel &model, float missing,
                                PredictionCacheEntry *out_preds,
                                uint32_t tree_begin, uint32_t tree_end) const {
    uint32_t const output_groups =  model.learner_model_param->num_output_group;

    auto m = dmlc::get<std::shared_ptr<Adapter>>(x);
    CHECK_EQ(m->NumColumns(), model.learner_model_param->num_feature)
        << "Number of columns in data must equal to trained model.";
    CHECK_EQ(dh::CurrentDevice(), m->DeviceIdx())
        << "XGBoost is running on device: " << this->ctx_->gpu_id << ", "
        << "but data is on: " << m->DeviceIdx();
    if (p_m) {
      p_m->Info().num_row_ = m->NumRows();
      this->InitOutPredictions(p_m->Info(), &(out_preds->predictions), model);
    } else {
      MetaInfo info;
      info.num_row_ = m->NumRows();
      this->InitOutPredictions(info, &(out_preds->predictions), model);
    }
    out_preds->predictions.SetDevice(m->DeviceIdx());

    const uint32_t BLOCK_THREADS = 128;
    auto GRID_SIZE = static_cast<uint32_t>(common::DivRoundUp(m->NumRows(), BLOCK_THREADS));

    auto max_shared_memory_bytes = dh::MaxSharedMemory(m->DeviceIdx());
    size_t shared_memory_bytes =
        SharedMemoryBytes<BLOCK_THREADS>(m->NumColumns(), max_shared_memory_bytes);
    DeviceModel d_model;
    d_model.Init(model, tree_begin, tree_end, m->DeviceIdx());

    bool use_shared = shared_memory_bytes != 0;
    size_t entry_start = 0;

    dh::LaunchKernel {GRID_SIZE, BLOCK_THREADS, shared_memory_bytes} (
        PredictKernel<Loader, typename Loader::BatchT>, m->Value(),
        d_model.nodes.ConstDeviceSpan(), out_preds->predictions.DeviceSpan(),
        d_model.tree_segments.ConstDeviceSpan(), d_model.tree_group.ConstDeviceSpan(),
        d_model.split_types.ConstDeviceSpan(),
        d_model.categories_tree_segments.ConstDeviceSpan(),
        d_model.categories_node_segments.ConstDeviceSpan(),
        d_model.categories.ConstDeviceSpan(), tree_begin, tree_end, m->NumColumns(),
        m->NumRows(), entry_start, use_shared, output_groups, missing);
  }

  bool InplacePredict(std::shared_ptr<DMatrix> p_m, const gbm::GBTreeModel& model, float missing,
                      PredictionCacheEntry* out_preds, uint32_t tree_begin,
                      unsigned tree_end) const override {
    auto proxy = dynamic_cast<data::DMatrixProxy*>(p_m.get());
    CHECK(proxy)<< "Inplace predict accepts only DMatrixProxy as input.";
    auto x = proxy->Adapter();
    if (x.type() == typeid(std::shared_ptr<data::CupyAdapter>)) {
      this->DispatchedInplacePredict<data::CupyAdapter,
                                     DeviceAdapterLoader<data::CupyAdapterBatch>>(
          x, p_m, model, missing, out_preds, tree_begin, tree_end);
    } else if (x.type() == typeid(std::shared_ptr<data::CudfAdapter>)) {
      this->DispatchedInplacePredict<data::CudfAdapter,
                                     DeviceAdapterLoader<data::CudfAdapterBatch>>(
          x, p_m, model, missing, out_preds, tree_begin, tree_end);
    } else {
      return false;
    }
    return true;
  }

  void PredictContribution(DMatrix* p_fmat,
                           HostDeviceVector<bst_float>* out_contribs,
                           const gbm::GBTreeModel& model, unsigned tree_end,
                           std::vector<bst_float> const* tree_weights,
                           bool approximate, int,
                           unsigned) const override {
    std::string not_implemented{"contribution is not implemented in GPU "
                                "predictor, use `cpu_predictor` instead."};
    if (approximate) {
      LOG(FATAL) << "Approximated " << not_implemented;
    }
    if (tree_weights != nullptr) {
      LOG(FATAL) << "Dart booster feature " << not_implemented;
    }
    dh::safe_cuda(hipSetDevice(ctx_->gpu_id));
    out_contribs->SetDevice(ctx_->gpu_id);
    if (tree_end == 0 || tree_end > model.trees.size()) {
      tree_end = static_cast<uint32_t>(model.trees.size());
    }

    const int ngroup = model.learner_model_param->num_output_group;
    CHECK_NE(ngroup, 0);
    // allocate space for (number of features + bias) times the number of rows
    size_t contributions_columns =
        model.learner_model_param->num_feature + 1;  // +1 for bias
    out_contribs->Resize(p_fmat->Info().num_row_ * contributions_columns *
                    model.learner_model_param->num_output_group);
    out_contribs->Fill(0.0f);
    auto phis = out_contribs->DeviceSpan();

    dh::device_vector<gpu_treeshap::PathElement<ShapSplitCondition>>
        device_paths;
    DeviceModel d_model;
    d_model.Init(model, 0, tree_end, ctx_->gpu_id);
    dh::device_vector<uint32_t> categories;
    ExtractPaths(&device_paths, &d_model, &categories, ctx_->gpu_id);
    for (auto& batch : p_fmat->GetBatches<SparsePage>()) {
      batch.data.SetDevice(ctx_->gpu_id);
      batch.offset.SetDevice(ctx_->gpu_id);
      SparsePageView X(batch.data.DeviceSpan(), batch.offset.DeviceSpan(),
                       model.learner_model_param->num_feature);
      auto begin = dh::tbegin(phis) + batch.base_rowid * contributions_columns;
      gpu_treeshap::GPUTreeShap<dh::XGBDeviceAllocator<int>>(
          X, device_paths.begin(), device_paths.end(), ngroup, begin,
          dh::tend(phis));
    }
    // Add the base margin term to last column
    p_fmat->Info().base_margin_.SetDevice(ctx_->gpu_id);
    const auto margin = p_fmat->Info().base_margin_.Data()->ConstDeviceSpan();
    float base_score = model.learner_model_param->base_score;
    dh::LaunchN(
        p_fmat->Info().num_row_ * model.learner_model_param->num_output_group,
        [=] __device__(size_t idx) {
          phis[(idx + 1) * contributions_columns - 1] +=
              margin.empty() ? base_score : margin[idx];
        });
  }

  void PredictInteractionContributions(DMatrix* p_fmat,
                                       HostDeviceVector<bst_float>* out_contribs,
                                       const gbm::GBTreeModel& model,
                                       unsigned tree_end,
                                       std::vector<bst_float> const* tree_weights,
                                       bool approximate) const override {
    std::string not_implemented{"contribution is not implemented in GPU "
                                "predictor, use `cpu_predictor` instead."};
    if (approximate) {
      LOG(FATAL) << "Approximated " << not_implemented;
    }
    if (tree_weights != nullptr) {
      LOG(FATAL) << "Dart booster feature " << not_implemented;
    }
    dh::safe_cuda(hipSetDevice(ctx_->gpu_id));
    out_contribs->SetDevice(ctx_->gpu_id);
    if (tree_end == 0 || tree_end > model.trees.size()) {
      tree_end = static_cast<uint32_t>(model.trees.size());
    }

    const int ngroup = model.learner_model_param->num_output_group;
    CHECK_NE(ngroup, 0);
    // allocate space for (number of features + bias) times the number of rows
    size_t contributions_columns =
        model.learner_model_param->num_feature + 1;  // +1 for bias
    out_contribs->Resize(p_fmat->Info().num_row_ * contributions_columns *
                         contributions_columns *
                         model.learner_model_param->num_output_group);
    out_contribs->Fill(0.0f);
    auto phis = out_contribs->DeviceSpan();

    dh::device_vector<gpu_treeshap::PathElement<ShapSplitCondition>>
        device_paths;
    DeviceModel d_model;
    d_model.Init(model, 0, tree_end, ctx_->gpu_id);
    dh::device_vector<uint32_t> categories;
    ExtractPaths(&device_paths, &d_model, &categories, ctx_->gpu_id);
    for (auto& batch : p_fmat->GetBatches<SparsePage>()) {
      batch.data.SetDevice(ctx_->gpu_id);
      batch.offset.SetDevice(ctx_->gpu_id);
      SparsePageView X(batch.data.DeviceSpan(), batch.offset.DeviceSpan(),
                       model.learner_model_param->num_feature);
      auto begin = dh::tbegin(phis) + batch.base_rowid * contributions_columns;
      gpu_treeshap::GPUTreeShapInteractions<dh::XGBDeviceAllocator<int>>(
          X, device_paths.begin(), device_paths.end(), ngroup, begin,
          dh::tend(phis));
    }
    // Add the base margin term to last column
    p_fmat->Info().base_margin_.SetDevice(ctx_->gpu_id);
    const auto margin = p_fmat->Info().base_margin_.Data()->ConstDeviceSpan();
    float base_score = model.learner_model_param->base_score;
    size_t n_features = model.learner_model_param->num_feature;
    dh::LaunchN(
        p_fmat->Info().num_row_ * model.learner_model_param->num_output_group,
        [=] __device__(size_t idx) {
          size_t group = idx % ngroup;
          size_t row_idx = idx / ngroup;
          phis[gpu_treeshap::IndexPhiInteractions(
              row_idx, ngroup, group, n_features, n_features, n_features)] +=
              margin.empty() ? base_score : margin[idx];
        });
  }

  void PredictInstance(const SparsePage::Inst&,
                       std::vector<bst_float>*,
                       const gbm::GBTreeModel&, unsigned) const override {
    LOG(FATAL) << "[Internal error]: " << __func__
               << " is not implemented in GPU Predictor.";
  }

  void PredictLeaf(DMatrix *p_fmat, HostDeviceVector<bst_float> *predictions,
                   const gbm::GBTreeModel &model,
                   unsigned tree_end) const override {
    dh::safe_cuda(hipSetDevice(ctx_->gpu_id));
    auto max_shared_memory_bytes = ConfigureDevice(ctx_->gpu_id);

    const MetaInfo& info = p_fmat->Info();
    constexpr uint32_t kBlockThreads = 128;
    size_t shared_memory_bytes = SharedMemoryBytes<kBlockThreads>(
        info.num_col_, max_shared_memory_bytes);
    bool use_shared = shared_memory_bytes != 0;
    bst_feature_t num_features = info.num_col_;
    bst_row_t num_rows = info.num_row_;
    size_t entry_start = 0;

    if (tree_end == 0 || tree_end > model.trees.size()) {
      tree_end = static_cast<uint32_t>(model.trees.size());
    }
    predictions->SetDevice(ctx_->gpu_id);
    predictions->Resize(num_rows * tree_end);
    DeviceModel d_model;
    d_model.Init(model, 0, tree_end, this->ctx_->gpu_id);

    if (p_fmat->PageExists<SparsePage>()) {
      for (auto const& batch : p_fmat->GetBatches<SparsePage>()) {
        batch.data.SetDevice(ctx_->gpu_id);
        batch.offset.SetDevice(ctx_->gpu_id);
        bst_row_t batch_offset = 0;
        SparsePageView data{batch.data.DeviceSpan(), batch.offset.DeviceSpan(),
                            model.learner_model_param->num_feature};
        size_t num_rows = batch.Size();
        auto grid =
            static_cast<uint32_t>(common::DivRoundUp(num_rows, kBlockThreads));
        dh::LaunchKernel {grid, kBlockThreads, shared_memory_bytes} (
            PredictLeafKernel<SparsePageLoader, SparsePageView>, data,
            d_model.nodes.ConstDeviceSpan(),
            predictions->DeviceSpan().subspan(batch_offset),
            d_model.tree_segments.ConstDeviceSpan(),

            d_model.split_types.ConstDeviceSpan(),
            d_model.categories_tree_segments.ConstDeviceSpan(),
            d_model.categories_node_segments.ConstDeviceSpan(),
            d_model.categories.ConstDeviceSpan(),

            d_model.tree_beg_, d_model.tree_end_, num_features, num_rows,
            entry_start, use_shared, nan(""));
        batch_offset += batch.Size();
      }
    } else {
      for (auto const& batch : p_fmat->GetBatches<EllpackPage>(BatchParam{})) {
        bst_row_t batch_offset = 0;
        EllpackDeviceAccessor data{batch.Impl()->GetDeviceAccessor(ctx_->gpu_id)};
        size_t num_rows = batch.Size();
        auto grid =
            static_cast<uint32_t>(common::DivRoundUp(num_rows, kBlockThreads));
        dh::LaunchKernel {grid, kBlockThreads, shared_memory_bytes} (
            PredictLeafKernel<EllpackLoader, EllpackDeviceAccessor>, data,
            d_model.nodes.ConstDeviceSpan(),
            predictions->DeviceSpan().subspan(batch_offset),
            d_model.tree_segments.ConstDeviceSpan(),

            d_model.split_types.ConstDeviceSpan(),
            d_model.categories_tree_segments.ConstDeviceSpan(),
            d_model.categories_node_segments.ConstDeviceSpan(),
            d_model.categories.ConstDeviceSpan(),

            d_model.tree_beg_, d_model.tree_end_, num_features, num_rows,
            entry_start, use_shared, nan(""));
        batch_offset += batch.Size();
      }
    }
  }

  void Configure(const std::vector<std::pair<std::string, std::string>>& cfg) override {
    Predictor::Configure(cfg);
  }

 private:
  /*! \brief Reconfigure the device when GPU is changed. */
  static size_t ConfigureDevice(int device) {
    if (device >= 0) {
      return dh::MaxSharedMemory(device);
    }
    return 0;
  }
};

XGBOOST_REGISTER_PREDICTOR(GPUPredictor, "gpu_predictor")
.describe("Make predictions using GPU.")
.set_body([](GenericParameter const* generic_param) {
            return new GPUPredictor(generic_param);
          });

}  // namespace predictor
}  // namespace xgboost
