#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2020 by Contributors
 */
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <GPUTreeShap/gpu_treeshap.h>
#include <memory>

#include "xgboost/data.h"
#include "xgboost/predictor.h"
#include "xgboost/tree_model.h"
#include "xgboost/tree_updater.h"
#include "xgboost/host_device_vector.h"

#include "../gbm/gbtree_model.h"
#include "../data/ellpack_page.cuh"
#include "../data/device_adapter.cuh"
#include "../common/common.h"
#include "../common/bitfield.h"
#include "../common/categorical.h"
#include "../common/device_helpers.cuh"

namespace xgboost {
namespace predictor {

DMLC_REGISTRY_FILE_TAG(gpu_predictor);

struct SparsePageView {
  common::Span<const Entry> d_data;
  common::Span<const bst_row_t> d_row_ptr;
  bst_feature_t num_features;

  SparsePageView() = default;
  XGBOOST_DEVICE SparsePageView(common::Span<const Entry> data,
                                common::Span<const bst_row_t> row_ptr,
                                bst_feature_t num_features)
      : d_data{data}, d_row_ptr{row_ptr}, num_features(num_features) {}
  __device__ float GetElement(size_t ridx, size_t fidx) const {
    // Binary search
    auto begin_ptr = d_data.begin() + d_row_ptr[ridx];
    auto end_ptr = d_data.begin() + d_row_ptr[ridx + 1];
    if (end_ptr - begin_ptr == this->NumCols()) {
      // Bypass span check for dense data
      return d_data.data()[d_row_ptr[ridx] + fidx].fvalue;
    }
    common::Span<const Entry>::iterator previous_middle;
    while (end_ptr != begin_ptr) {
      auto middle = begin_ptr + (end_ptr - begin_ptr) / 2;
      if (middle == previous_middle) {
        break;
      } else {
        previous_middle = middle;
      }

      if (middle->index == fidx) {
        return middle->fvalue;
      } else if (middle->index < fidx) {
        begin_ptr = middle;
      } else {
        end_ptr = middle;
      }
    }
    // Value is missing
    return nanf("");
  }
  XGBOOST_DEVICE size_t NumRows() const { return d_row_ptr.size() - 1; }
  XGBOOST_DEVICE size_t NumCols() const { return num_features; }
};

struct SparsePageLoader {
  bool use_shared;
  SparsePageView data;
  float* smem;
  size_t entry_start;

  __device__ SparsePageLoader(SparsePageView data, bool use_shared, bst_feature_t num_features,
                              bst_row_t num_rows, size_t entry_start)
      : use_shared(use_shared),
        data(data),
        entry_start(entry_start) {
    extern __shared__ float _smem[];
    smem = _smem;
    // Copy instances
    if (use_shared) {
      bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
      int shared_elements = blockDim.x * data.num_features;
      dh::BlockFill(smem, shared_elements, nanf(""));
      __syncthreads();
      if (global_idx < num_rows) {
        bst_uint elem_begin = data.d_row_ptr[global_idx];
        bst_uint elem_end = data.d_row_ptr[global_idx + 1];
        for (bst_uint elem_idx = elem_begin; elem_idx < elem_end; elem_idx++) {
          Entry elem = data.d_data[elem_idx - entry_start];
          smem[threadIdx.x * data.num_features + elem.index] = elem.fvalue;
        }
      }
      __syncthreads();
    }
  }
  __device__ float GetElement(size_t  ridx, size_t  fidx) const {
    if (use_shared) {
      return smem[threadIdx.x * data.num_features + fidx];
    } else {
      return data.GetElement(ridx, fidx);
    }
  }
};

struct EllpackLoader {
  EllpackDeviceAccessor const& matrix;
  XGBOOST_DEVICE EllpackLoader(EllpackDeviceAccessor const& m, bool,
                               bst_feature_t, bst_row_t, size_t)
      : matrix{m} {}
  __device__ __forceinline__ float GetElement(size_t  ridx, size_t  fidx) const {
    auto gidx = matrix.GetBinIndex(ridx, fidx);
    if (gidx == -1) {
      return nan("");
    }
    // The gradient index needs to be shifted by one as min values are not included in the
    // cuts.
    if (gidx == matrix.feature_segments[fidx]) {
      return matrix.min_fvalue[fidx];
    }
    return matrix.gidx_fvalue_map[gidx - 1];
  }
};

template <typename Batch>
struct DeviceAdapterLoader {
  Batch batch;
  bst_feature_t columns;
  float* smem;
  bool use_shared;

  using BatchT = Batch;

  XGBOOST_DEV_INLINE DeviceAdapterLoader(Batch const batch, bool use_shared,
                                         bst_feature_t num_features, bst_row_t num_rows,
                                         size_t entry_start) :
    batch{batch},
    columns{num_features},
    use_shared{use_shared} {
      extern __shared__ float _smem[];
      smem = _smem;
      if (use_shared) {
        uint32_t global_idx = blockDim.x * blockIdx.x + threadIdx.x;
        size_t shared_elements = blockDim.x * num_features;
        dh::BlockFill(smem, shared_elements, nanf(""));
        __syncthreads();
        if (global_idx < num_rows) {
          auto beg = global_idx * columns;
          auto end = (global_idx + 1) * columns;
          for (size_t i = beg; i < end; ++i) {
            smem[threadIdx.x * num_features + (i - beg)] = batch.GetElement(i).value;
          }
        }
      }
      __syncthreads();
    }

  XGBOOST_DEV_INLINE  float GetElement(size_t  ridx, size_t  fidx) const {
    if (use_shared) {
      return smem[threadIdx.x * columns + fidx];
    }
    return batch.GetElement(ridx * columns + fidx).value;
  }
};

template <typename Loader>
__device__ float GetLeafWeight(bst_row_t ridx, const RegTree::Node* tree,
                               common::Span<FeatureType const> split_types,
                               common::Span<RegTree::Segment const> d_cat_ptrs,
                               common::Span<uint32_t const> d_categories,
                               Loader* loader) {
  bst_node_t nidx = 0;
  RegTree::Node n = tree[nidx];
  while (!n.IsLeaf()) {
    float fvalue = loader->GetElement(ridx, n.SplitIndex());
    // Missing value
    if (common::CheckNAN(fvalue)) {
      nidx = n.DefaultChild();
    } else {
      bool go_left = true;
      if (common::IsCat(split_types, nidx)) {
        auto categories = d_categories.subspan(d_cat_ptrs[nidx].beg,
                                               d_cat_ptrs[nidx].size);
        go_left = Decision(categories, common::AsCat(fvalue));
      } else {
        go_left = fvalue < n.SplitCond();
      }
      if (go_left) {
        nidx = n.LeftChild();
      } else {
        nidx = n.RightChild();
      }
    }
    n = tree[nidx];
  }
  return tree[nidx].LeafValue();
}

template <typename Loader>
__device__ bst_node_t GetLeafIndex(bst_row_t ridx, const RegTree::Node* tree,
                                   Loader const& loader) {
  bst_node_t nidx = 0;
  RegTree::Node n = tree[nidx];
  while (!n.IsLeaf()) {
    float fvalue = loader.GetElement(ridx, n.SplitIndex());
    // Missing value
    if (isnan(fvalue)) {
      nidx = n.DefaultChild();
      n = tree[nidx];
    } else {
      if (fvalue < n.SplitCond()) {
        nidx = n.LeftChild();
        n = tree[nidx];
      } else {
        nidx = n.RightChild();
        n = tree[nidx];
      }
    }
  }
  return nidx;
}

template <typename Loader, typename Data>
__global__ void PredictLeafKernel(Data data,
                                  common::Span<const RegTree::Node> d_nodes,
                                  common::Span<float> d_out_predictions,
                                  common::Span<size_t const> d_tree_segments,
                                  size_t tree_begin, size_t tree_end, size_t num_features,
                                  size_t num_rows, size_t entry_start, bool use_shared) {
  bst_row_t ridx = blockDim.x * blockIdx.x + threadIdx.x;
  if (ridx >= num_rows) {
    return;
  }
  Loader loader(data, use_shared, num_features, num_rows, entry_start);
  for (int tree_idx = tree_begin; tree_idx < tree_end; ++tree_idx) {
    const RegTree::Node* d_tree = &d_nodes[d_tree_segments[tree_idx - tree_begin]];
    auto leaf = GetLeafIndex(ridx, d_tree, loader);
    d_out_predictions[ridx * (tree_end - tree_begin) + tree_idx] = leaf;
  }
}

template <typename Loader, typename Data>
__global__ void
PredictKernel(Data data, common::Span<const RegTree::Node> d_nodes,
              common::Span<float> d_out_predictions,
              common::Span<size_t const> d_tree_segments,
              common::Span<int const> d_tree_group,
              common::Span<FeatureType const> d_tree_split_types,
              common::Span<uint32_t const> d_cat_tree_segments,
              common::Span<RegTree::Segment const> d_cat_node_segments,
              common::Span<uint32_t const> d_categories, size_t tree_begin,
              size_t tree_end, size_t num_features, size_t num_rows,
              size_t entry_start, bool use_shared, int num_group) {
  bst_uint global_idx = blockDim.x * blockIdx.x + threadIdx.x;
  Loader loader(data, use_shared, num_features, num_rows, entry_start);
  if (global_idx >= num_rows) return;
  if (num_group == 1) {
    float sum = 0;
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      const RegTree::Node* d_tree =
          &d_nodes[d_tree_segments[tree_idx - tree_begin]];
      auto tree_cat_ptrs = d_cat_node_segments.subspan(
          d_tree_segments[tree_idx - tree_begin],
          d_tree_segments[tree_idx - tree_begin + 1] -
              d_tree_segments[tree_idx - tree_begin]);
      auto tree_categories =
          d_categories.subspan(d_cat_tree_segments[tree_idx - tree_begin],
                               d_cat_tree_segments[tree_idx - tree_begin + 1] -
                               d_cat_tree_segments[tree_idx - tree_begin]);
      float leaf = GetLeafWeight(global_idx, d_tree, d_tree_split_types,
                                 tree_cat_ptrs,
                                 tree_categories,
                                 &loader);
      sum += leaf;
    }
    d_out_predictions[global_idx] += sum;
  } else {
    for (int tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      int tree_group = d_tree_group[tree_idx];
      const RegTree::Node* d_tree =
          &d_nodes[d_tree_segments[tree_idx - tree_begin]];
      bst_uint out_prediction_idx = global_idx * num_group + tree_group;
      auto tree_cat_ptrs = d_cat_node_segments.subspan(
          d_tree_segments[tree_idx - tree_begin],
          d_tree_segments[tree_idx - tree_begin + 1] -
              d_tree_segments[tree_idx - tree_begin]);
      auto tree_categories =
          d_categories.subspan(d_cat_tree_segments[tree_idx - tree_begin],
                               d_cat_tree_segments[tree_idx - tree_begin + 1] -
                               d_cat_tree_segments[tree_idx - tree_begin]);
      d_out_predictions[out_prediction_idx] +=
          GetLeafWeight(global_idx, d_tree, d_tree_split_types,
                        tree_cat_ptrs,
                        tree_categories,
                        &loader);
    }
  }
}

class DeviceModel {
 public:
  // Need to lazily construct the vectors because GPU id is only known at runtime
  HostDeviceVector<RTreeNodeStat> stats;
  HostDeviceVector<size_t> tree_segments;
  HostDeviceVector<RegTree::Node> nodes;
  HostDeviceVector<int> tree_group;
  HostDeviceVector<FeatureType> split_types;

  // Pointer to each tree, segmenting the node array.
  HostDeviceVector<uint32_t> categories_tree_segments;
  // Pointer to each node, segmenting categories array.
  HostDeviceVector<RegTree::Segment> categories_node_segments;
  HostDeviceVector<uint32_t> categories;

  size_t tree_beg_;  // NOLINT
  size_t tree_end_;  // NOLINT
  int num_group;

  void Init(const gbm::GBTreeModel& model, size_t tree_begin, size_t tree_end, int32_t gpu_id) {
    dh::safe_cuda(hipSetDevice(gpu_id));

    CHECK_EQ(model.param.size_leaf_vector, 0);
    // Copy decision trees to device
    tree_segments = std::move(HostDeviceVector<size_t>({}, gpu_id));
    auto& h_tree_segments = tree_segments.HostVector();
    h_tree_segments.reserve((tree_end - tree_begin) + 1);
    size_t sum = 0;
    h_tree_segments.push_back(sum);
    for (auto tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      sum += model.trees.at(tree_idx)->GetNodes().size();
      h_tree_segments.push_back(sum);
    }

    nodes = std::move(HostDeviceVector<RegTree::Node>(h_tree_segments.back(), RegTree::Node(),
                                                      gpu_id));
    stats = std::move(HostDeviceVector<RTreeNodeStat>(h_tree_segments.back(),
                                                      RTreeNodeStat(), gpu_id));
    auto d_nodes = nodes.DevicePointer();
    auto d_stats = stats.DevicePointer();
    for (auto tree_idx = tree_begin; tree_idx < tree_end; tree_idx++) {
      auto& src_nodes = model.trees.at(tree_idx)->GetNodes();
      auto& src_stats = model.trees.at(tree_idx)->GetStats();
      dh::safe_cuda(hipMemcpyAsync(
          d_nodes + h_tree_segments[tree_idx - tree_begin], src_nodes.data(),
          sizeof(RegTree::Node) * src_nodes.size(), hipMemcpyDefault));
      dh::safe_cuda(hipMemcpyAsync(
          d_stats + h_tree_segments[tree_idx - tree_begin], src_stats.data(),
          sizeof(RTreeNodeStat) * src_stats.size(), hipMemcpyDefault));
    }

    tree_group = std::move(HostDeviceVector<int>(model.tree_info.size(), 0, gpu_id));
    auto& h_tree_group = tree_group.HostVector();
    std::memcpy(h_tree_group.data(), model.tree_info.data(), sizeof(int) * model.tree_info.size());

    // Initialize categorical splits.
    split_types.SetDevice(gpu_id);
    std::vector<FeatureType>& h_split_types = split_types.HostVector();
    h_split_types.resize(h_tree_segments.back());
    for (auto tree_idx = tree_begin; tree_idx < tree_end; ++tree_idx) {
      auto const& src_st = model.trees.at(tree_idx)->GetSplitTypes();
      std::copy(src_st.cbegin(), src_st.cend(),
                h_split_types.begin() + h_tree_segments[tree_idx - tree_begin]);
    }

    categories = HostDeviceVector<uint32_t>({}, gpu_id);
    categories_tree_segments = HostDeviceVector<uint32_t>(1, 0, gpu_id);
    std::vector<uint32_t> &h_categories = categories.HostVector();
    std::vector<uint32_t> &h_split_cat_segments = categories_tree_segments.HostVector();
    for (auto tree_idx = tree_begin; tree_idx < tree_end; ++tree_idx) {
      auto const& src_cats = model.trees.at(tree_idx)->GetSplitCategories();
      size_t orig_size = h_categories.size();
      h_categories.resize(orig_size + src_cats.size());
      std::copy(src_cats.cbegin(), src_cats.cend(),
                h_categories.begin() + orig_size);
      h_split_cat_segments.push_back(h_categories.size());
    }

    categories_node_segments =
        HostDeviceVector<RegTree::Segment>(h_tree_segments.back(), {}, gpu_id);
    std::vector<RegTree::Segment> &h_categories_node_segments =
        categories_node_segments.HostVector();
    for (auto tree_idx = tree_begin; tree_idx < tree_end; ++tree_idx) {
      auto const &src_cats_ptr = model.trees.at(tree_idx)->GetSplitCategoriesPtr();
      std::copy(src_cats_ptr.cbegin(), src_cats_ptr.cend(),
                h_categories_node_segments.begin() +
                    h_tree_segments[tree_idx - tree_begin]);
    }

    this->tree_beg_ = tree_begin;
    this->tree_end_ = tree_end;
    this->num_group = model.learner_model_param->num_output_group;
  }
};

struct PathInfo {
  int64_t leaf_position;  // -1 not a leaf
  size_t length;
  size_t tree_idx;
};

// Transform model into path element form for GPUTreeShap
void ExtractPaths(dh::device_vector<gpu_treeshap::PathElement>* paths,
                  const gbm::GBTreeModel& model, size_t tree_limit,
                  int gpu_id) {
  DeviceModel device_model;
  device_model.Init(model, 0, tree_limit, gpu_id);
  dh::caching_device_vector<PathInfo> info(device_model.nodes.Size());
  dh::XGBCachingDeviceAllocator<PathInfo> alloc;
  auto d_nodes = device_model.nodes.ConstDeviceSpan();
  auto d_tree_segments = device_model.tree_segments.ConstDeviceSpan();
  auto nodes_transform = dh::MakeTransformIterator<PathInfo>(
      thrust::make_counting_iterator(0ull), [=] __device__(size_t idx) {
        auto n = d_nodes[idx];
        if (!n.IsLeaf() || n.IsDeleted()) {
          return PathInfo{-1, 0, 0};
        }
        size_t tree_idx =
            dh::SegmentId(d_tree_segments.begin(), d_tree_segments.end(), idx);
        size_t tree_offset = d_tree_segments[tree_idx];
        size_t path_length = 1;
        while (!n.IsRoot()) {
          n = d_nodes[n.Parent() + tree_offset];
          path_length++;
        }
        return PathInfo{int64_t(idx), path_length, tree_idx};
      });
  auto end = thrust::copy_if(
      thrust::cuda::par(alloc), nodes_transform,
      nodes_transform + d_nodes.size(), info.begin(),
      [=] __device__(const PathInfo& e) { return e.leaf_position != -1; });
  info.resize(end - info.begin());
  auto length_iterator = dh::MakeTransformIterator<size_t>(
      info.begin(),
      [=] __device__(const PathInfo& info) { return info.length; });
  dh::caching_device_vector<size_t> path_segments(info.size() + 1);
  thrust::exclusive_scan(thrust::cuda::par(alloc), length_iterator,
                         length_iterator + info.size() + 1,
                         path_segments.begin());

  paths->resize(path_segments.back());

  auto d_paths = paths->data().get();
  auto d_info = info.data().get();
  auto d_stats = device_model.stats.ConstDeviceSpan();
  auto d_tree_group = device_model.tree_group.ConstDeviceSpan();
  auto d_path_segments = path_segments.data().get();
  dh::LaunchN(gpu_id, info.size(), [=] __device__(size_t idx) {
    auto path_info = d_info[idx];
    size_t tree_offset = d_tree_segments[path_info.tree_idx];
    int group = d_tree_group[path_info.tree_idx];
    size_t child_idx = path_info.leaf_position;
    auto child = d_nodes[child_idx];
    float v = child.LeafValue();
    const float inf = std::numeric_limits<float>::infinity();
    size_t output_position = d_path_segments[idx + 1] - 1;
    while (!child.IsRoot()) {
      size_t parent_idx = tree_offset + child.Parent();
      double child_cover = d_stats[child_idx].sum_hess;
      double parent_cover = d_stats[parent_idx].sum_hess;
      double zero_fraction = child_cover / parent_cover;
      auto parent = d_nodes[parent_idx];
      bool is_left_path = (tree_offset + parent.LeftChild()) == child_idx;
      bool is_missing_path = (!parent.DefaultLeft() && !is_left_path) ||
                             (parent.DefaultLeft() && is_left_path);
      float lower_bound = is_left_path ? -inf : parent.SplitCond();
      float upper_bound = is_left_path ? parent.SplitCond() : inf;
      d_paths[output_position--] = {
          idx,         parent.SplitIndex(), group,         lower_bound,
          upper_bound, is_missing_path,     zero_fraction, v};
      child_idx = parent_idx;
      child = parent;
    }
    // Root node has feature -1
    d_paths[output_position] = {idx, -1, group, -inf, inf, false, 1.0, v};
  });
}

namespace {
template <size_t kBlockThreads>
size_t SharedMemoryBytes(size_t cols, size_t max_shared_memory_bytes) {
  // No way max_shared_memory_bytes that is equal to 0.
  CHECK_GT(max_shared_memory_bytes, 0);
  size_t shared_memory_bytes =
      static_cast<size_t>(sizeof(float) * cols * kBlockThreads);
  if (shared_memory_bytes > max_shared_memory_bytes) {
    shared_memory_bytes = 0;
  }
  return shared_memory_bytes;
}
}  // anonymous namespace

class GPUPredictor : public xgboost::Predictor {
 private:
  void PredictInternal(const SparsePage& batch,
                       size_t num_features,
                       HostDeviceVector<bst_float>* predictions,
                       size_t batch_offset) {
    batch.offset.SetDevice(generic_param_->gpu_id);
    batch.data.SetDevice(generic_param_->gpu_id);
    const uint32_t BLOCK_THREADS = 128;
    size_t num_rows = batch.Size();
    auto GRID_SIZE = static_cast<uint32_t>(common::DivRoundUp(num_rows, BLOCK_THREADS));

    size_t shared_memory_bytes =
        SharedMemoryBytes<BLOCK_THREADS>(num_features, max_shared_memory_bytes_);
    bool use_shared = shared_memory_bytes != 0;

    size_t entry_start = 0;
    SparsePageView data(batch.data.DeviceSpan(), batch.offset.DeviceSpan(),
                        num_features);
    dh::LaunchKernel {GRID_SIZE, BLOCK_THREADS, shared_memory_bytes} (
        PredictKernel<SparsePageLoader, SparsePageView>, data,
        model_.nodes.ConstDeviceSpan(),
        predictions->DeviceSpan().subspan(batch_offset),
        model_.tree_segments.ConstDeviceSpan(), model_.tree_group.ConstDeviceSpan(),
        model_.split_types.ConstDeviceSpan(),
        model_.categories_tree_segments.ConstDeviceSpan(),
        model_.categories_node_segments.ConstDeviceSpan(),
        model_.categories.ConstDeviceSpan(), model_.tree_beg_, model_.tree_end_,
        num_features, num_rows, entry_start, use_shared, model_.num_group);
  }
  void PredictInternal(EllpackDeviceAccessor const& batch,
                       HostDeviceVector<bst_float>* out_preds,
                       size_t batch_offset) {
    const uint32_t BLOCK_THREADS = 256;
    size_t num_rows = batch.n_rows;
    auto GRID_SIZE = static_cast<uint32_t>(common::DivRoundUp(num_rows, BLOCK_THREADS));

    bool use_shared = false;
    size_t entry_start = 0;
    dh::LaunchKernel {GRID_SIZE, BLOCK_THREADS} (
        PredictKernel<EllpackLoader, EllpackDeviceAccessor>, batch,
        model_.nodes.ConstDeviceSpan(), out_preds->DeviceSpan().subspan(batch_offset),
        model_.tree_segments.ConstDeviceSpan(), model_.tree_group.ConstDeviceSpan(),
        model_.split_types.ConstDeviceSpan(),
        model_.categories_tree_segments.ConstDeviceSpan(),
        model_.categories_node_segments.ConstDeviceSpan(),
        model_.categories.ConstDeviceSpan(), model_.tree_beg_, model_.tree_end_,
        batch.NumFeatures(), num_rows, entry_start, use_shared,
        model_.num_group);
  }

  void DevicePredictInternal(DMatrix* dmat, HostDeviceVector<float>* out_preds,
                             const gbm::GBTreeModel& model, size_t tree_begin,
                             size_t tree_end) {
    dh::safe_cuda(hipSetDevice(generic_param_->gpu_id));
    if (tree_end - tree_begin == 0) {
      return;
    }
    model_.Init(model, tree_begin, tree_end, generic_param_->gpu_id);
    out_preds->SetDevice(generic_param_->gpu_id);
    auto const& info = dmat->Info();

    if (dmat->PageExists<SparsePage>()) {
      size_t batch_offset = 0;
      for (auto &batch : dmat->GetBatches<SparsePage>()) {
        this->PredictInternal(batch, model.learner_model_param->num_feature,
                              out_preds, batch_offset);
        batch_offset += batch.Size() * model.learner_model_param->num_output_group;
      }
    } else {
      size_t batch_offset = 0;
      for (auto const& page : dmat->GetBatches<EllpackPage>()) {
        this->PredictInternal(
            page.Impl()->GetDeviceAccessor(generic_param_->gpu_id),
            out_preds,
            batch_offset);
        batch_offset += page.Impl()->n_rows;
      }
    }
  }

 public:
  explicit GPUPredictor(GenericParameter const* generic_param) :
      Predictor::Predictor{generic_param} {}

  ~GPUPredictor() override {
    if (generic_param_->gpu_id >= 0) {
      dh::safe_cuda(hipSetDevice(generic_param_->gpu_id));
    }
  }

  void PredictBatch(DMatrix* dmat, PredictionCacheEntry* predts,
                    const gbm::GBTreeModel& model, int tree_begin,
                    unsigned ntree_limit = 0) override {
    // This function is duplicated with CPU predictor PredictBatch, see comments in there.
    // FIXME(trivialfis): Remove the duplication.
    std::lock_guard<std::mutex> const guard(lock_);
    int device = generic_param_->gpu_id;
    CHECK_GE(device, 0) << "Set `gpu_id' to positive value for processing GPU data.";
    ConfigureDevice(device);

    CHECK_EQ(tree_begin, 0);
    auto* out_preds = &predts->predictions;
    CHECK_GE(predts->version, tree_begin);
    if (out_preds->Size() == 0 && dmat->Info().num_row_ != 0) {
      CHECK_EQ(predts->version, 0);
    }
    if (predts->version == 0) {
      this->InitOutPredictions(dmat->Info(), out_preds, model);
    }

    uint32_t const output_groups =  model.learner_model_param->num_output_group;
    CHECK_NE(output_groups, 0);

    uint32_t real_ntree_limit = ntree_limit * output_groups;
    if (real_ntree_limit == 0 || real_ntree_limit > model.trees.size()) {
      real_ntree_limit = static_cast<uint32_t>(model.trees.size());
    }

    uint32_t const end_version = (tree_begin + real_ntree_limit) / output_groups;

    if (predts->version > end_version) {
      CHECK_NE(ntree_limit, 0);
      this->InitOutPredictions(dmat->Info(), out_preds, model);
      predts->version = 0;
    }
    uint32_t const beg_version = predts->version;
    CHECK_LE(beg_version, end_version);

    if (beg_version < end_version) {
      this->DevicePredictInternal(dmat, out_preds, model,
                                  beg_version * output_groups,
                                  end_version * output_groups);
    }

    uint32_t delta = end_version - beg_version;
    CHECK_LE(delta, model.trees.size());
    predts->Update(delta);

    CHECK(out_preds->Size() == output_groups * dmat->Info().num_row_ ||
          out_preds->Size() == dmat->Info().num_row_);
  }

  template <typename Adapter, typename Loader>
  void DispatchedInplacePredict(dmlc::any const &x,
                                const gbm::GBTreeModel &model, float,
                                PredictionCacheEntry *out_preds,
                                uint32_t tree_begin, uint32_t tree_end) const {
    auto max_shared_memory_bytes = dh::MaxSharedMemory(this->generic_param_->gpu_id);
    uint32_t const output_groups =  model.learner_model_param->num_output_group;
    DeviceModel d_model;
    d_model.Init(model, tree_begin, tree_end, this->generic_param_->gpu_id);

    auto m = dmlc::get<std::shared_ptr<Adapter>>(x);
    CHECK_EQ(m->NumColumns(), model.learner_model_param->num_feature)
        << "Number of columns in data must equal to trained model.";
    CHECK_EQ(this->generic_param_->gpu_id, m->DeviceIdx())
        << "XGBoost is running on device: " << this->generic_param_->gpu_id << ", "
        << "but data is on: " << m->DeviceIdx();
    MetaInfo info;
    info.num_col_ = m->NumColumns();
    info.num_row_ = m->NumRows();
    this->InitOutPredictions(info, &(out_preds->predictions), model);

    const uint32_t BLOCK_THREADS = 128;
    auto GRID_SIZE = static_cast<uint32_t>(common::DivRoundUp(info.num_row_, BLOCK_THREADS));

    size_t shared_memory_bytes =
        SharedMemoryBytes<BLOCK_THREADS>(info.num_col_, max_shared_memory_bytes);
    bool use_shared = shared_memory_bytes != 0;
    size_t entry_start = 0;

    dh::LaunchKernel {GRID_SIZE, BLOCK_THREADS, shared_memory_bytes} (
        PredictKernel<Loader, typename Loader::BatchT>, m->Value(),
        d_model.nodes.ConstDeviceSpan(), out_preds->predictions.DeviceSpan(),
        d_model.tree_segments.ConstDeviceSpan(), d_model.tree_group.ConstDeviceSpan(),
        d_model.split_types.ConstDeviceSpan(),
        d_model.categories_tree_segments.ConstDeviceSpan(),
        d_model.categories_node_segments.ConstDeviceSpan(),
        d_model.categories.ConstDeviceSpan(), tree_begin, tree_end, m->NumColumns(),
        info.num_row_, entry_start, use_shared, output_groups);
  }

  void InplacePredict(dmlc::any const &x, const gbm::GBTreeModel &model,
                      float missing, PredictionCacheEntry *out_preds,
                      uint32_t tree_begin, unsigned tree_end) const override {
    if (x.type() == typeid(std::shared_ptr<data::CupyAdapter>)) {
      this->DispatchedInplacePredict<
          data::CupyAdapter, DeviceAdapterLoader<data::CupyAdapterBatch>>(
          x, model, missing, out_preds, tree_begin, tree_end);
    } else if (x.type() == typeid(std::shared_ptr<data::CudfAdapter>)) {
      this->DispatchedInplacePredict<
          data::CudfAdapter, DeviceAdapterLoader<data::CudfAdapterBatch>>(
          x, model, missing, out_preds, tree_begin, tree_end);
    } else {
      LOG(FATAL) << "Only CuPy and CuDF are supported by GPU Predictor.";
    }
  }

  void PredictContribution(DMatrix* p_fmat,
                           HostDeviceVector<bst_float>* out_contribs,
                           const gbm::GBTreeModel& model, unsigned ntree_limit,
                           std::vector<bst_float>*,
                           bool approximate, int,
                           unsigned) override {
    if (approximate) {
      LOG(FATAL) << "Approximated contribution is not implemented in GPU Predictor.";
    }

    dh::safe_cuda(hipSetDevice(generic_param_->gpu_id));
    out_contribs->SetDevice(generic_param_->gpu_id);
    uint32_t real_ntree_limit =
        ntree_limit * model.learner_model_param->num_output_group;
    if (real_ntree_limit == 0 || real_ntree_limit > model.trees.size()) {
      real_ntree_limit = static_cast<uint32_t>(model.trees.size());
    }

    const int ngroup = model.learner_model_param->num_output_group;
    CHECK_NE(ngroup, 0);
    // allocate space for (number of features + bias) times the number of rows
    size_t contributions_columns =
        model.learner_model_param->num_feature + 1;  // +1 for bias
    out_contribs->Resize(p_fmat->Info().num_row_ * contributions_columns *
                    model.learner_model_param->num_output_group);
    out_contribs->Fill(0.0f);
    auto phis = out_contribs->DeviceSpan();

    dh::device_vector<gpu_treeshap::PathElement> device_paths;
    ExtractPaths(&device_paths, model, real_ntree_limit,
                 generic_param_->gpu_id);
    for (auto& batch : p_fmat->GetBatches<SparsePage>()) {
      batch.data.SetDevice(generic_param_->gpu_id);
      batch.offset.SetDevice(generic_param_->gpu_id);
      SparsePageView X(batch.data.DeviceSpan(), batch.offset.DeviceSpan(),
                       model.learner_model_param->num_feature);
      gpu_treeshap::GPUTreeShap(
          X, device_paths.begin(), device_paths.end(), ngroup,
          phis.data() + batch.base_rowid * contributions_columns, phis.size());
    }
    // Add the base margin term to last column
    p_fmat->Info().base_margin_.SetDevice(generic_param_->gpu_id);
    const auto margin = p_fmat->Info().base_margin_.ConstDeviceSpan();
    float base_score = model.learner_model_param->base_score;
    dh::LaunchN(
        generic_param_->gpu_id,
        p_fmat->Info().num_row_ * model.learner_model_param->num_output_group,
        [=] __device__(size_t idx) {
          phis[(idx + 1) * contributions_columns - 1] +=
              margin.empty() ? base_score : margin[idx];
        });
  }

  void PredictInteractionContributions(DMatrix* p_fmat,
                                       HostDeviceVector<bst_float>* out_contribs,
                                       const gbm::GBTreeModel& model,
                                       unsigned ntree_limit,
                                       std::vector<bst_float>*,
                                       bool approximate) override {
    if (approximate) {
      LOG(FATAL) << "[Internal error]: " << __func__
                 << " approximate is not implemented in GPU Predictor.";
    }

    dh::safe_cuda(hipSetDevice(generic_param_->gpu_id));
    out_contribs->SetDevice(generic_param_->gpu_id);
    uint32_t real_ntree_limit =
        ntree_limit * model.learner_model_param->num_output_group;
    if (real_ntree_limit == 0 || real_ntree_limit > model.trees.size()) {
      real_ntree_limit = static_cast<uint32_t>(model.trees.size());
    }

    const int ngroup = model.learner_model_param->num_output_group;
    CHECK_NE(ngroup, 0);
    // allocate space for (number of features + bias) times the number of rows
    size_t contributions_columns =
        model.learner_model_param->num_feature + 1;  // +1 for bias
    out_contribs->Resize(p_fmat->Info().num_row_ * contributions_columns *
                         contributions_columns *
                         model.learner_model_param->num_output_group);
    out_contribs->Fill(0.0f);
    auto phis = out_contribs->DeviceSpan();

    dh::device_vector<gpu_treeshap::PathElement> device_paths;
    ExtractPaths(&device_paths, model, real_ntree_limit,
                 generic_param_->gpu_id);
    for (auto& batch : p_fmat->GetBatches<SparsePage>()) {
      batch.data.SetDevice(generic_param_->gpu_id);
      batch.offset.SetDevice(generic_param_->gpu_id);
      SparsePageView X(batch.data.DeviceSpan(), batch.offset.DeviceSpan(),
                       model.learner_model_param->num_feature);
      gpu_treeshap::GPUTreeShapInteractions(
          X, device_paths.begin(), device_paths.end(), ngroup,
          phis.data() + batch.base_rowid * contributions_columns, phis.size());
    }
    // Add the base margin term to last column
    p_fmat->Info().base_margin_.SetDevice(generic_param_->gpu_id);
    const auto margin = p_fmat->Info().base_margin_.ConstDeviceSpan();
    float base_score = model.learner_model_param->base_score;
    size_t n_features = model.learner_model_param->num_feature;
    dh::LaunchN(
        generic_param_->gpu_id,
        p_fmat->Info().num_row_ * model.learner_model_param->num_output_group,
        [=] __device__(size_t idx) {
          size_t group = idx % ngroup;
          size_t row_idx = idx / ngroup;
          phis[gpu_treeshap::IndexPhiInteractions(
              row_idx, ngroup, group, n_features, n_features, n_features)] +=
              margin.empty() ? base_score : margin[idx];
        });
  }

 protected:
  void InitOutPredictions(const MetaInfo& info,
                          HostDeviceVector<bst_float>* out_preds,
                          const gbm::GBTreeModel& model) const {
    size_t n_classes = model.learner_model_param->num_output_group;
    size_t n = n_classes * info.num_row_;
    const HostDeviceVector<bst_float>& base_margin = info.base_margin_;
    out_preds->SetDevice(generic_param_->gpu_id);
    out_preds->Resize(n);
    if (base_margin.Size() != 0) {
      CHECK_EQ(base_margin.Size(), n);
      out_preds->Copy(base_margin);
    } else {
      out_preds->Fill(model.learner_model_param->base_score);
    }
  }

  void PredictInstance(const SparsePage::Inst&,
                       std::vector<bst_float>*,
                       const gbm::GBTreeModel&, unsigned) override {
    LOG(FATAL) << "[Internal error]: " << __func__
               << " is not implemented in GPU Predictor.";
  }

  void PredictLeaf(DMatrix* p_fmat, HostDeviceVector<bst_float>* predictions,
                   const gbm::GBTreeModel& model,
                   unsigned ntree_limit) override {
    dh::safe_cuda(hipSetDevice(generic_param_->gpu_id));
    ConfigureDevice(generic_param_->gpu_id);

    const MetaInfo& info = p_fmat->Info();
    constexpr uint32_t kBlockThreads = 128;
    size_t shared_memory_bytes =
        SharedMemoryBytes<kBlockThreads>(info.num_col_, max_shared_memory_bytes_);
    bool use_shared = shared_memory_bytes != 0;
    bst_feature_t num_features = info.num_col_;
    bst_row_t num_rows = info.num_row_;
    size_t entry_start = 0;

    uint32_t real_ntree_limit = ntree_limit * model.learner_model_param->num_output_group;
    if (real_ntree_limit == 0 || real_ntree_limit > model.trees.size()) {
      real_ntree_limit = static_cast<uint32_t>(model.trees.size());
    }
    predictions->SetDevice(generic_param_->gpu_id);
    predictions->Resize(num_rows * real_ntree_limit);
    model_.Init(model, 0, real_ntree_limit, generic_param_->gpu_id);

    if (p_fmat->PageExists<SparsePage>()) {
      for (auto const& batch : p_fmat->GetBatches<SparsePage>()) {
        batch.data.SetDevice(generic_param_->gpu_id);
        batch.offset.SetDevice(generic_param_->gpu_id);
        bst_row_t batch_offset = 0;
        SparsePageView data{batch.data.DeviceSpan(), batch.offset.DeviceSpan(),
                            model.learner_model_param->num_feature};
        size_t num_rows = batch.Size();
        auto grid =
            static_cast<uint32_t>(common::DivRoundUp(num_rows, kBlockThreads));
        dh::LaunchKernel {grid, kBlockThreads, shared_memory_bytes} (
            PredictLeafKernel<SparsePageLoader, SparsePageView>, data,
            model_.nodes.ConstDeviceSpan(),
            predictions->DeviceSpan().subspan(batch_offset),
            model_.tree_segments.ConstDeviceSpan(),
            model_.tree_beg_, model_.tree_end_, num_features, num_rows,
            entry_start, use_shared);
        batch_offset += batch.Size();
      }
    } else {
      for (auto const& batch : p_fmat->GetBatches<EllpackPage>()) {
        bst_row_t batch_offset = 0;
        EllpackDeviceAccessor data{batch.Impl()->GetDeviceAccessor(generic_param_->gpu_id)};
        size_t num_rows = batch.Size();
        auto grid =
            static_cast<uint32_t>(common::DivRoundUp(num_rows, kBlockThreads));
        dh::LaunchKernel {grid, kBlockThreads, shared_memory_bytes} (
            PredictLeafKernel<EllpackLoader, EllpackDeviceAccessor>, data,
            model_.nodes.ConstDeviceSpan(),
            predictions->DeviceSpan().subspan(batch_offset),
            model_.tree_segments.ConstDeviceSpan(),
            model_.tree_beg_, model_.tree_end_, num_features, num_rows,
            entry_start, use_shared);
        batch_offset += batch.Size();
      }
    }
  }

  void Configure(const std::vector<std::pair<std::string, std::string>>& cfg) override {
    Predictor::Configure(cfg);
  }

 private:
  /*! \brief Reconfigure the device when GPU is changed. */
  void ConfigureDevice(int device) {
    if (device >= 0) {
      max_shared_memory_bytes_ = dh::MaxSharedMemory(device);
    }
  }

  std::mutex lock_;
  DeviceModel model_;
  size_t max_shared_memory_bytes_ { 0 };
};

XGBOOST_REGISTER_PREDICTOR(GPUPredictor, "gpu_predictor")
.describe("Make predictions using GPU.")
.set_body([](GenericParameter const* generic_param) {
            return new GPUPredictor(generic_param);
          });

}  // namespace predictor
}  // namespace xgboost
