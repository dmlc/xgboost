#include "hip/hip_runtime.h"
/*!
 * Copyright 2020 by Contributors
 * \file rank_metric.cc
 * \brief prediction rank based metrics.
 * \author Kailong Chen, Tianqi Chen
 */
#include <rabit/rabit.h>
#include <dmlc/registry.h>

#include <xgboost/metric.h>
#include <xgboost/host_device_vector.h>
#include <thrust/iterator/discard_iterator.h>

#include <cmath>
#include <vector>

#include "metric_common.h"

#include "../common/math.h"
#include "../common/device_helpers.cuh"

namespace xgboost {
namespace metric {
// tag the this file, used by force static link later.
DMLC_REGISTRY_FILE_TAG(rank_metric_gpu);

/*! \brief Evaluate rank list on GPU */
template <typename EvalMetricT>
struct EvalRankGpu : public Metric, public EvalRankConfig {
 public:
  bst_float Eval(const HostDeviceVector<bst_float> &preds,
                 const MetaInfo &info,
                 bool distributed) override {
    // Sanity check is done by the caller
    std::vector<unsigned> tgptr(2, 0);
    tgptr[1] = static_cast<unsigned>(preds.Size());
    const std::vector<unsigned> &gptr = info.group_ptr_.size() == 0 ? tgptr : info.group_ptr_;

    const auto ngroups = static_cast<bst_omp_uint>(gptr.size() - 1);

    auto device = tparam_->gpu_id;
    dh::safe_cuda(hipSetDevice(device));

    info.labels_.SetDevice(device);
    preds.SetDevice(device);

    auto dpreds = preds.ConstDevicePointer();
    auto dlabels = info.labels_.ConstDevicePointer();

    // Sort all the predictions
    dh::SegmentSorter<float> segment_pred_sorter;
    segment_pred_sorter.SortItems(dpreds, preds.Size(), gptr);

    // Compute individual group metric and sum them up
    return EvalMetricT::EvalMetric(segment_pred_sorter, dlabels, *this);
  }

  const char* Name() const override {
    return name.c_str();
  }

  explicit EvalRankGpu(const char* name, const char* param) {
    using namespace std;  // NOLINT(*)
    if (param != nullptr) {
      std::ostringstream os;
      if (sscanf(param, "%u[-]?", &this->topn) == 1) {
        os << name << '@' << param;
        this->name = os.str();
      } else {
        os << name << param;
        this->name = os.str();
      }
      if (param[strlen(param) - 1] == '-') {
        this->minus = true;
      }
    } else {
      this->name = name;
    }
  }
};

/*! \brief Precision at N, for both classification and rank */
struct EvalPrecisionGpu {
 public:
  static double EvalMetric(const dh::SegmentSorter<float> &pred_sorter,
                           const float *dlabels,
                           const EvalRankConfig &ecfg) {
    // Group info on device
    const auto &dgroups = pred_sorter.GetGroupsSpan();
    const auto ngroups = pred_sorter.GetNumGroups();
    const auto &dgroup_idx = pred_sorter.GetGroupSegmentsSpan();

    // Original positions of the predictions after they have been sorted
    const auto &dpreds_orig_pos = pred_sorter.GetOriginalPositionsSpan();

    // First, determine non zero labels in the dataset individually
    auto DetermineNonTrivialLabelLambda = [=] __device__(uint32_t idx) {
      return (static_cast<unsigned>(dlabels[dpreds_orig_pos[idx]]) != 0) ? 1 : 0;
    };  // NOLINT

    // Find each group's metric sum
    dh::caching_device_vector<uint32_t> hits(ngroups, 0);
    const auto nitems = pred_sorter.GetNumItems();
    auto *dhits = hits.data().get();

    int device_id = -1;
    dh::safe_cuda(hipGetDevice(&device_id));
    // For each group item compute the aggregated precision
    dh::LaunchN(device_id, nitems, nullptr, [=] __device__(uint32_t idx) {
      const auto group_idx = dgroup_idx[idx];
      const auto group_begin = dgroups[group_idx];
      const auto ridx = idx - group_begin;
      if (ridx < ecfg.topn && DetermineNonTrivialLabelLambda(idx)) {
        atomicAdd(&dhits[group_idx], 1);
      }
    });

    // Allocator to be used for managing space overhead while performing reductions
    dh::XGBCachingDeviceAllocator<char> alloc;
    return static_cast<double>(thrust::reduce(thrust::cuda::par(alloc),
                                              hits.begin(), hits.end())) / ecfg.topn;
  }
};

/*! \brief NDCG: Normalized Discounted Cumulative Gain at N */
struct EvalNDCGGpu {
 public:
  static void ComputeDCG(const dh::SegmentSorter<float> &pred_sorter,
                         const float *dlabels,
                         const EvalRankConfig &ecfg,
                         // The order in which labels have to be accessed. The order is determined
                         // by sorting the predictions or the labels for the entire dataset
                         const xgboost::common::Span<const uint32_t> &dlabels_sort_order,
                         dh::caching_device_vector<double> *dcgptr) {
    dh::caching_device_vector<double> &dcgs(*dcgptr);
    // Group info on device
    const auto &dgroups = pred_sorter.GetGroupsSpan();
    const auto &dgroup_idx = pred_sorter.GetGroupSegmentsSpan();

    // First, determine non zero labels in the dataset individually
    auto DetermineNonTrivialLabelLambda = [=] __device__(uint32_t idx) {
      return (static_cast<unsigned>(dlabels[dlabels_sort_order[idx]]));
    };  // NOLINT

    // Find each group's DCG value
    const auto nitems = pred_sorter.GetNumItems();
    auto *ddcgs = dcgs.data().get();

    int device_id = -1;
    dh::safe_cuda(hipGetDevice(&device_id));

    // For each group item compute the aggregated precision
    dh::LaunchN(device_id, nitems, nullptr, [=] __device__(uint32_t idx) {
      const auto group_idx = dgroup_idx[idx];
      const auto group_begin = dgroups[group_idx];
      const auto ridx = idx - group_begin;
      auto label = DetermineNonTrivialLabelLambda(idx);
      if (ridx < ecfg.topn && label) {
        atomicAdd(&ddcgs[group_idx], ((1 << label) - 1) / std::log2(ridx + 2.0));
      }
    });
  }

  static double EvalMetric(const dh::SegmentSorter<float> &pred_sorter,
                           const float *dlabels,
                           const EvalRankConfig &ecfg) {
    // Sort the labels and compute IDCG
    dh::SegmentSorter<float> segment_label_sorter;
    segment_label_sorter.SortItems(dlabels, pred_sorter.GetNumItems(),
                                   pred_sorter.GetGroupSegmentsSpan());

    uint32_t ngroups = pred_sorter.GetNumGroups();

    dh::caching_device_vector<double> idcg(ngroups, 0);
    ComputeDCG(pred_sorter, dlabels, ecfg, segment_label_sorter.GetOriginalPositionsSpan(), &idcg);

    // Compute the DCG values next
    dh::caching_device_vector<double> dcg(ngroups, 0);
    ComputeDCG(pred_sorter, dlabels, ecfg, pred_sorter.GetOriginalPositionsSpan(), &dcg);

    double *ddcg = dcg.data().get();
    double *didcg = idcg.data().get();

    int device_id = -1;
    dh::safe_cuda(hipGetDevice(&device_id));
    // Compute the group's DCG and reduce it across all groups
    dh::LaunchN(device_id, ngroups, nullptr, [=] __device__(uint32_t gidx) {
      if (didcg[gidx] == 0.0f) {
        ddcg[gidx] = (ecfg.minus) ? 0.0f : 1.0f;
      } else {
        ddcg[gidx] /= didcg[gidx];
      }
    });

    // Allocator to be used for managing space overhead while performing reductions
    dh::XGBCachingDeviceAllocator<char> alloc;
    return thrust::reduce(thrust::cuda::par(alloc), dcg.begin(), dcg.end());
  }
};

/*! \brief Mean Average Precision at N, for both classification and rank */
struct EvalMAPGpu {
 public:
  static double EvalMetric(const dh::SegmentSorter<float> &pred_sorter,
                           const float *dlabels,
                           const EvalRankConfig &ecfg) {
    // Group info on device
    const auto &dgroups = pred_sorter.GetGroupsSpan();
    const auto ngroups = pred_sorter.GetNumGroups();
    const auto &dgroup_idx = pred_sorter.GetGroupSegmentsSpan();

    // Original positions of the predictions after they have been sorted
    const auto &dpreds_orig_pos = pred_sorter.GetOriginalPositionsSpan();

    // First, determine non zero labels in the dataset individually
    const auto nitems = pred_sorter.GetNumItems();
    dh::caching_device_vector<uint32_t> hits(nitems, 0);
    auto DetermineNonTrivialLabelLambda = [=] __device__(uint32_t idx) {
      return (static_cast<unsigned>(dlabels[dpreds_orig_pos[idx]]) != 0) ? 1 : 0;
    };  // NOLINT

    thrust::transform(thrust::make_counting_iterator(static_cast<uint32_t>(0)),
                      thrust::make_counting_iterator(nitems),
                      hits.begin(),
                      DetermineNonTrivialLabelLambda);

    // Allocator to be used by sort for managing space overhead while performing prefix scans
    dh::XGBCachingDeviceAllocator<char> alloc;

    // Next, prefix scan the nontrivial labels that are segmented to accumulate them.
    // This is required for computing the metric sum
    // Data segmented into different groups...
    thrust::inclusive_scan_by_key(thrust::cuda::par(alloc),
                                  dh::tcbegin(dgroup_idx), dh::tcend(dgroup_idx),
                                  hits.begin(),  // Input value
                                  hits.begin());  // In-place scan

    // Find each group's metric sum
    dh::caching_device_vector<double> sumap(ngroups, 0);
    auto *dsumap = sumap.data().get();
    const auto *dhits = hits.data().get();

    int device_id = -1;
    dh::safe_cuda(hipGetDevice(&device_id));
    // For each group item compute the aggregated precision
    dh::LaunchN(device_id, nitems, nullptr, [=] __device__(uint32_t idx) {
      if (DetermineNonTrivialLabelLambda(idx)) {
        const auto group_idx = dgroup_idx[idx];
        const auto group_begin = dgroups[group_idx];
        const auto ridx = idx - group_begin;
        if (ridx < ecfg.topn) {
          atomicAdd(&dsumap[group_idx],
                    static_cast<double>(dhits[idx]) / (ridx + 1));
        }
      }
    });

    // Aggregate the group's item precisions
    dh::LaunchN(device_id, ngroups, nullptr, [=] __device__(uint32_t gidx) {
      auto nhits = dgroups[gidx + 1] ? dhits[dgroups[gidx + 1] - 1] : 0;
      if (nhits != 0) {
        dsumap[gidx] /= nhits;
      } else {
        if (ecfg.minus) {
          dsumap[gidx] = 0;
        } else {
          dsumap[gidx] = 1;
        }
      }
    });

    return thrust::reduce(thrust::cuda::par(alloc), sumap.begin(), sumap.end());
  }
};

/*! \brief Area Under Curve metric computation for ranking datasets */
struct EvalAucGpu : public Metric {
 public:
  // This function object computes the positive precision pair for each prediction group
  class ComputePosPair : public thrust::unary_function<uint32_t, double> {
   public:
    XGBOOST_DEVICE ComputePosPair(const double *pred_group_pos_precision,
                                  const double *pred_group_neg_precision,
                                  const double *pred_group_incr_precision)
      : pred_group_pos_precision_(pred_group_pos_precision),
        pred_group_neg_precision_(pred_group_neg_precision),
        pred_group_incr_precision_(pred_group_incr_precision) {}

    // Compute positive precision pair for the prediction group at 'idx'
    __device__ __forceinline__ double operator()(uint32_t idx) const {
        return pred_group_neg_precision_[idx] *
               (pred_group_incr_precision_[idx] + pred_group_pos_precision_[idx] * 0.5);
    }

   private:
    // Accumulated positive precision for the prediction group
    const double *pred_group_pos_precision_{nullptr};
    // Accumulated negative precision for the prediction group
    const double *pred_group_neg_precision_{nullptr};
    // Incremental positive precision for the prediction group
    const double *pred_group_incr_precision_{nullptr};
  };

  template <typename T>
  void ReleaseMemory(dh::caching_device_vector<T> &vec) {  // NOLINT
    dh::caching_device_vector<T>().swap(vec);
  }

  bst_float Eval(const HostDeviceVector<bst_float> &preds,
                 const MetaInfo &info,
                 bool distributed) override {
    // Sanity check is done by the caller
    std::vector<unsigned> tgptr(2, 0);
    tgptr[1] = static_cast<unsigned>(info.labels_.Size());
    const std::vector<unsigned> &gptr = info.group_ptr_.empty() ? tgptr : info.group_ptr_;

    auto device = tparam_->gpu_id;
    dh::safe_cuda(hipSetDevice(device));

    info.labels_.SetDevice(device);
    preds.SetDevice(device);
    info.weights_.SetDevice(device);

    auto dpreds = preds.ConstDevicePointer();
    auto dlabels = info.labels_.ConstDevicePointer();
    auto dweights = info.weights_.ConstDevicePointer();

    // Sort all the predictions (from one or more groups)
    dh::SegmentSorter<float> segment_pred_sorter;
    segment_pred_sorter.SortItems(dpreds, preds.Size(), gptr);

    const auto &dsorted_preds = segment_pred_sorter.GetItemsSpan();
    const auto &dpreds_orig_pos = segment_pred_sorter.GetOriginalPositionsSpan();

    // Group info on device
    const auto &dgroups = segment_pred_sorter.GetGroupsSpan();
    uint32_t ngroups = segment_pred_sorter.GetNumGroups();

    // Final values
    double hsum_auc = 0.0;
    unsigned hauc_error = 0;

    int device_id = -1;
    dh::safe_cuda(hipGetDevice(&device_id));

    // Allocator to be used for managing space overhead while performing reductions
    dh::XGBCachingDeviceAllocator<char> alloc;

    if (ngroups == 1) {
      const auto nitems = segment_pred_sorter.GetNumItems();

      // First, segment all the predictions in the group. This is required so that we can
      // aggregate the positive and negative precisions within that prediction group
      dh::caching_device_vector<unsigned> dpred_segs(nitems, 0);
      auto *pred_seg_arr = dpred_segs.data().get();
      // This is for getting the next segment number
      dh::caching_device_vector<unsigned> seg_idx(1, 0);
      auto *seg_idx_ptr = seg_idx.data().get();

      dh::caching_device_vector<double> dbuf_pos(nitems, 0);
      dh::caching_device_vector<double> dbuf_neg(nitems, 0);
      auto *buf_pos_arr = dbuf_pos.data().get();
      auto *buf_neg_arr = dbuf_neg.data().get();

      dh::LaunchN(device_id, nitems, nullptr, [=] __device__(int idx) {
        auto ctr = dlabels[dpreds_orig_pos[idx]];
        // For ranking task, weights are per-group
        // For binary classification task, weights are per-instance
        const auto wt = dweights == nullptr ? 1.0f : dweights[dpreds_orig_pos[idx]];
        buf_pos_arr[idx] = ctr * wt;
        buf_neg_arr[idx] = (1.0f - ctr) * wt;
        if (idx == nitems - 1 || dsorted_preds[idx] != dsorted_preds[idx + 1]) {
          auto new_seg_idx = atomicAdd(seg_idx_ptr, 1);
          auto pred_val = dsorted_preds[idx];
          do {
            pred_seg_arr[idx] = new_seg_idx;
            idx--;
          } while (idx >= 0 && dsorted_preds[idx] == pred_val);
        }
      });

      auto nunique_preds = seg_idx.back();
      ReleaseMemory(seg_idx);

      // Next, accumulate the positive and negative precisions for every prediction group
      dh::caching_device_vector<double> sum_dbuf_pos(nunique_preds, 0);
      auto itr = thrust::reduce_by_key(thrust::cuda::par(alloc),
                                       dpred_segs.begin(), dpred_segs.end(),  // Segmented by this
                                       dbuf_pos.begin(),  // Individual precisions
                                       thrust::make_discard_iterator(),  // Ignore unique segments
                                       sum_dbuf_pos.begin());  // Write accumulated results here
      ReleaseMemory(dbuf_pos);
      CHECK(itr.second - sum_dbuf_pos.begin() == nunique_preds);

      dh::caching_device_vector<double> sum_dbuf_neg(nunique_preds, 0);
      itr = thrust::reduce_by_key(thrust::cuda::par(alloc),
                                  dpred_segs.begin(), dpred_segs.end(),
                                  dbuf_neg.begin(),
                                  thrust::make_discard_iterator(),
                                  sum_dbuf_neg.begin());
      ReleaseMemory(dbuf_neg);
      ReleaseMemory(dpred_segs);
      CHECK(itr.second - sum_dbuf_neg.begin() == nunique_preds);

      dh::caching_device_vector<double> sum_nneg(nunique_preds, 0);
      thrust::inclusive_scan(thrust::cuda::par(alloc),
                             sum_dbuf_neg.begin(), sum_dbuf_neg.end(),
                             sum_nneg.begin());
      double sum_neg_prec_val = sum_nneg.back();
      ReleaseMemory(sum_nneg);

      // Find incremental sum for the positive precisions that is then used to
      // compute incremental positive precision pair
      dh::caching_device_vector<double> sum_npos(nunique_preds + 1, 0);
      thrust::inclusive_scan(thrust::cuda::par(alloc),
                             sum_dbuf_pos.begin(), sum_dbuf_pos.end(),
                             sum_npos.begin() + 1);
      double sum_pos_prec_val = sum_npos.back();

      if (sum_pos_prec_val <= 0.0 || sum_neg_prec_val <= 0.0) {
        hauc_error = 1;
      } else {
        dh::caching_device_vector<double> sum_pospair(nunique_preds, 0);
        // Finally, compute the positive precision pair
        thrust::transform(thrust::make_counting_iterator(static_cast<uint32_t>(0)),
                          thrust::make_counting_iterator(static_cast<uint32_t>(nunique_preds)),
                          sum_pospair.begin(),
                          ComputePosPair(sum_dbuf_pos.data().get(),
                                         sum_dbuf_neg.data().get(),
                                         sum_npos.data().get()));
        ReleaseMemory(sum_dbuf_pos);
        ReleaseMemory(sum_dbuf_neg);
        ReleaseMemory(sum_npos);
        hsum_auc = thrust::reduce(thrust::cuda::par(alloc),
                                  sum_pospair.begin(), sum_pospair.end())
                     / (sum_pos_prec_val * sum_neg_prec_val);
      }
    } else {
      // AUC sum for each group
      dh::caching_device_vector<double> sum_auc(ngroups, 0);
      // AUC error across all groups
      dh::caching_device_vector<int> auc_error(1, 0);
      auto *dsum_auc = sum_auc.data().get();
      auto *dauc_error = auc_error.data().get();

      // For each group item compute the aggregated precision
      dh::LaunchN<1, 32>(device_id, ngroups, nullptr, [=] __device__(uint32_t gidx) {
        double sum_pospair = 0.0, sum_npos = 0.0, sum_nneg = 0.0, buf_pos = 0.0, buf_neg = 0.0;

        for (auto i = dgroups[gidx]; i < dgroups[gidx + 1]; ++i) {
          const auto ctr = dlabels[dpreds_orig_pos[i]];
          // Keep bucketing predictions in same bucket
          if (i != dgroups[gidx] && dsorted_preds[i] != dsorted_preds[i - 1]) {
            sum_pospair += buf_neg * (sum_npos + buf_pos * 0.5);
            sum_npos += buf_pos;
            sum_nneg += buf_neg;
            buf_neg = buf_pos = 0.0f;
          }
          // For ranking task, weights are per-group
          // For binary classification task, weights are per-instance
          const auto wt = dweights == nullptr ? 1.0f : dweights[gidx];
          buf_pos += ctr * wt;
          buf_neg += (1.0f - ctr) * wt;
        }
        sum_pospair += buf_neg * (sum_npos + buf_pos * 0.5);
        sum_npos += buf_pos;
        sum_nneg += buf_neg;

        // Check weird conditions
        if (sum_npos <= 0.0 || sum_nneg <= 0.0) {
          atomicAdd(dauc_error, 1);
        } else {
          // This is the AUC
          dsum_auc[gidx] = sum_pospair / (sum_npos * sum_nneg);
        }
      });

      hsum_auc = thrust::reduce(thrust::cuda::par(alloc), sum_auc.begin(), sum_auc.end());
      hauc_error = auc_error.back();  // Copy it back to host
    }

    // Report average AUC across all groups
    // In distributed mode, workers which only contains pos or neg samples
    // will be ignored when aggregate AUC.
    bst_float dat[2] = {0.0f, 0.0f};
    if (hauc_error < ngroups) {
      dat[0] = static_cast<bst_float>(hsum_auc);
      dat[1] = static_cast<bst_float>(ngroups - hauc_error);
    }
    if (distributed) {
      rabit::Allreduce<rabit::op::Sum>(dat, 2);
    }
    CHECK_GT(dat[1], 0.0f)
      << "AUC: the dataset only contains pos or neg samples";
    return dat[0] / dat[1];
  }

  const char* Name() const override {
    return "auc";
  }
};

/*! \brief Area Under PR Curve metric computation for ranking datasets */
struct EvalAucPRGpu : public Metric {
 public:
  // This function object computes the item's positive/negative precision value
  class ComputeItemPrecision : public thrust::unary_function<uint32_t, float> {
   public:
    // The precision type to be computed
    enum class PrecisionType {
      kPositive,
      kNegative
    };

    XGBOOST_DEVICE ComputeItemPrecision(PrecisionType ptype,
                                        uint32_t ngroups,
                                        const float *dweights,
                                        const xgboost::common::Span<const uint32_t> &dgidxs,
                                        const float *dlabels)
      : ptype_(ptype), ngroups_(ngroups), dweights_(dweights), dgidxs_(dgidxs), dlabels_(dlabels) {}

    // Compute precision value for the prediction that was originally at 'idx'
    __device__ __forceinline__ float operator()(uint32_t idx) const {
      // For ranking task, weights are per-group
      // For binary classification task, weights are per-instance
      const auto wt = dweights_ == nullptr ? 1.0f : dweights_[ngroups_ == 1 ? idx : dgidxs_[idx]];
      return wt * (ptype_ == PrecisionType::kPositive ? dlabels_[idx] : (1.0f - dlabels_[idx]));
    }

   private:
    PrecisionType ptype_;  // Precision type to be computed
    uint32_t ngroups_;  // Number of groups in the dataset
    const float *dweights_;  // Instance/group weights
    const xgboost::common::Span<const uint32_t> dgidxs_;  // The group a given instance belongs to
    const float *dlabels_;  // Unsorted labels in the dataset
  };

  bst_float Eval(const HostDeviceVector<bst_float> &preds,
                 const MetaInfo &info,
                 bool distributed) override {
    // Sanity check is done by the caller
    std::vector<unsigned> tgptr(2, 0);
    tgptr[1] = static_cast<unsigned>(info.labels_.Size());
    const std::vector<unsigned> &gptr = info.group_ptr_.empty() ? tgptr : info.group_ptr_;

    auto device = tparam_->gpu_id;
    dh::safe_cuda(hipSetDevice(device));

    info.labels_.SetDevice(device);
    preds.SetDevice(device);
    info.weights_.SetDevice(device);

    auto dpreds = preds.ConstDevicePointer();
    auto dlabels = info.labels_.ConstDevicePointer();
    auto dweights = info.weights_.ConstDevicePointer();

    // Sort all the predictions
    dh::SegmentSorter<float> segment_pred_sorter;
    segment_pred_sorter.SortItems(dpreds, preds.Size(), gptr);

    const auto &dsorted_preds = segment_pred_sorter.GetItemsSpan();
    // Original positions of the predictions after they have been sorted
    const auto &dpreds_orig_pos = segment_pred_sorter.GetOriginalPositionsSpan();

    // Group info on device
    const auto &dgroups = segment_pred_sorter.GetGroupsSpan();
    uint32_t ngroups = segment_pred_sorter.GetNumGroups();
    const auto &dgroup_idx = segment_pred_sorter.GetGroupSegmentsSpan();

    // First, aggregate the positive and negative precision for each group
    dh::caching_device_vector<double> total_pos(ngroups, 0);
    dh::caching_device_vector<double> total_neg(ngroups, 0);

    // Allocator to be used for managing space overhead while performing transformed reductions
    dh::XGBCachingDeviceAllocator<char> alloc;

    // Compute each elements positive precision value and reduce them across groups concurrently.
    ComputeItemPrecision pos_prec_functor(ComputeItemPrecision::PrecisionType::kPositive,
                                          ngroups, dweights, dgroup_idx, dlabels);
    auto end_range =
      thrust::reduce_by_key(thrust::cuda::par(alloc),
                            dh::tcbegin(dgroup_idx), dh::tcend(dgroup_idx),
                            thrust::make_transform_iterator(
                              // The indices need not be sequential within a group, as we care only
                              // about the sum of positive precision values within a group
                              dh::tcbegin(segment_pred_sorter.GetOriginalPositionsSpan()),
                              pos_prec_functor),
                            thrust::make_discard_iterator(),  // We don't care for the group indices
                            total_pos.begin());  // Sum of positive precision values in the group
    CHECK(end_range.second - total_pos.begin() == total_pos.size());

    // Compute each elements negative precision value and reduce them across groups concurrently.
    ComputeItemPrecision neg_prec_functor(ComputeItemPrecision::PrecisionType::kNegative,
                                          ngroups, dweights, dgroup_idx, dlabels);
    end_range =
      thrust::reduce_by_key(thrust::cuda::par(alloc),
                            dh::tcbegin(dgroup_idx), dh::tcend(dgroup_idx),
                            thrust::make_transform_iterator(
                              // The indices need not be sequential within a group, as we care only
                              // about the sum of negative precision values within a group
                              dh::tcbegin(segment_pred_sorter.GetOriginalPositionsSpan()),
                              neg_prec_functor),
                            thrust::make_discard_iterator(),  // We don't care for the group indices
                            total_neg.begin());  // Sum of negative precision values in the group
    CHECK(end_range.second - total_neg.begin() == total_neg.size());

    const auto *dtotal_pos = total_pos.data().get();
    const auto *dtotal_neg = total_neg.data().get();

    // AUC sum for each group
    dh::caching_device_vector<double> sum_auc(ngroups, 0);
    // AUC error across all groups
    dh::caching_device_vector<int> auc_error(1, 0);
    auto *dsum_auc = sum_auc.data().get();
    auto *dauc_error = auc_error.data().get();

    int device_id = -1;
    dh::safe_cuda(hipGetDevice(&device_id));
    // For each group item compute the aggregated precision
    dh::LaunchN<1, 32>(device_id, ngroups, nullptr, [=] __device__(uint32_t gidx) {
      // We need pos > 0 && neg > 0
      if (dtotal_pos[gidx] <= 0.0 || dtotal_neg[gidx] <= 0.0) {
        atomicAdd(dauc_error, 1);
      } else {
        auto gbegin = dgroups[gidx];
        auto gend = dgroups[gidx + 1];
        // Calculate AUC
        double tp = 0.0, prevtp = 0.0, fp = 0.0, prevfp = 0.0, h = 0.0, a = 0.0, b = 0.0;
        for (auto i = gbegin; i < gend; ++i) {
          const auto wt = dweights == nullptr ? 1.0f
                                              : dweights[ngroups == 1 ? dpreds_orig_pos[i] : gidx];
          tp += wt * dlabels[dpreds_orig_pos[i]];
          fp += wt * (1.0f - dlabels[dpreds_orig_pos[i]]);
          if ((i < gend - 1 && dsorted_preds[i] != dsorted_preds[i + 1]) || (i == gend - 1)) {
            if (tp == prevtp) {
              a = 1.0;
              b = 0.0;
            } else {
              h = (fp - prevfp) / (tp - prevtp);
              a = 1.0 + h;
              b = (prevfp - h * prevtp) / dtotal_pos[gidx];
            }
            if (0.0 != b) {
              dsum_auc[gidx] += (tp / dtotal_pos[gidx] - prevtp / dtotal_pos[gidx] -
                                 b / a * (std::log(a * tp / dtotal_pos[gidx] + b) -
                                          std::log(a * prevtp / dtotal_pos[gidx] + b))) / a;
            } else {
              dsum_auc[gidx] += (tp / dtotal_pos[gidx] - prevtp / dtotal_pos[gidx]) / a;
            }
            prevtp = tp;
            prevfp = fp;
          }
        }

        // Sanity check
        if (tp < 0 || prevtp < 0 || fp < 0 || prevfp < 0) {
          // Check if we have any metric error thus far
          auto current_auc_error = atomicAdd(dauc_error, 0);
          KERNEL_CHECK(!current_auc_error);
        }
      }
    });

    const auto hsum_auc = thrust::reduce(thrust::cuda::par(alloc), sum_auc.begin(), sum_auc.end());
    const auto hauc_error = auc_error.back();  // Copy it back to host

    // Report average AUC-PR across all groups
    // In distributed mode, workers which only contains pos or neg samples
    // will be ignored when aggregate AUC-PR.
    bst_float dat[2] = {0.0f, 0.0f};
    if (hauc_error < static_cast<int>(ngroups)) {
      dat[0] = static_cast<bst_float>(hsum_auc);
      dat[1] = static_cast<bst_float>(static_cast<int>(ngroups) - hauc_error);
    }
    if (distributed) {
      rabit::Allreduce<rabit::op::Sum>(dat, 2);
    }
    CHECK_GT(dat[1], 0.0f)
      << "AUC-PR: the dataset only contains pos or neg samples";
    CHECK_LE(dat[0], dat[1]) << "AUC-PR: AUC > 1.0";
    return dat[0] / dat[1];
  }

  const char* Name() const override {
    return "aucpr";
  }
};

XGBOOST_REGISTER_GPU_METRIC(AucGpu, "auc")
.describe("Area under curve for rank computed on GPU.")
.set_body([](const char* param) { return new EvalAucGpu(); });

XGBOOST_REGISTER_GPU_METRIC(AucPRGpu, "aucpr")
.describe("Area under PR curve for rank computed on GPU.")
.set_body([](const char* param) { return new EvalAucPRGpu(); });

XGBOOST_REGISTER_GPU_METRIC(PrecisionGpu, "pre")
.describe("precision@k for rank computed on GPU.")
.set_body([](const char* param) { return new EvalRankGpu<EvalPrecisionGpu>("pre", param); });

XGBOOST_REGISTER_GPU_METRIC(NDCGGpu, "ndcg")
.describe("ndcg@k for rank computed on GPU.")
.set_body([](const char* param) { return new EvalRankGpu<EvalNDCGGpu>("ndcg", param); });

XGBOOST_REGISTER_GPU_METRIC(MAPGpu, "map")
.describe("map@k for rank computed on GPU.")
.set_body([](const char* param) { return new EvalRankGpu<EvalMAPGpu>("map", param); });
}  // namespace metric
}  // namespace xgboost
