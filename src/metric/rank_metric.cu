#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2023 by XGBoost Contributors
 */
#include <dmlc/registry.h>
#include <thrust/iterator/counting_iterator.h>  // for make_counting_iterator
#include <thrust/reduce.h>                      // for reduce

#include <algorithm>                            // for transform
#include <cstddef>                              // for size_t
#include <memory>                               // for shared_ptr
#include <vector>                               // for vector

#include "../common/cuda_context.cuh"           // for HIPContext
#include "../common/device_helpers.cuh"         // for MakeTransformIterator
#include "../common/optional_weight.h"          // for MakeOptionalWeights
#include "../common/ranking_utils.cuh"          // for CalcQueriesDCG, NDCGCache
#include "metric_common.h"
#include "rank_metric.h"
#include "xgboost/base.h"                // for XGBOOST_DEVICE
#include "xgboost/context.h"             // for Context
#include "xgboost/data.h"                // for MetaInfo
#include "xgboost/host_device_vector.h"  // for HostDeviceVector
#include "xgboost/linalg.h"              // for MakeTensorView
#include "xgboost/logging.h"             // for CHECK
#include "xgboost/metric.h"

namespace xgboost::metric {
// tag the this file, used by force static link later.
DMLC_REGISTRY_FILE_TAG(rank_metric_gpu);

namespace cuda_impl {
PackedReduceResult PreScore(Context const *ctx, MetaInfo const &info,
                            HostDeviceVector<float> const &predt,
                            std::shared_ptr<ltr::PreCache> p_cache) {
  auto d_gptr = p_cache->DataGroupPtr(ctx);
  auto d_label = info.labels.View(ctx->gpu_id).Slice(linalg::All(), 0);

  predt.SetDevice(ctx->gpu_id);
  auto d_rank_idx = p_cache->SortedIdx(ctx, predt.ConstDeviceSpan());
  auto topk = p_cache->Param().TopK();
  auto d_weight = common::MakeOptionalWeights(ctx, info.weights_);

  auto it = dh::MakeTransformIterator<double>(
      thrust::make_counting_iterator(0ul), [=] XGBOOST_DEVICE(std::size_t i) {
        auto g = dh::SegmentId(d_gptr, i);
        auto g_begin = d_gptr[g];
        auto g_end = d_gptr[g + 1];
        i -= g_begin;
        auto g_label = d_label.Slice(linalg::Range(g_begin, g_end));
        auto g_rank = d_rank_idx.subspan(g_begin, g_end - g_begin);
        double y = g_label(g_rank[i]);
        auto n = std::min(static_cast<std::size_t>(topk), g_label.Size());
        double w{d_weight[g]};
        if (i >= n) {
          return 0.0;
        }
        return y / static_cast<double>(n) * w;
      });

  auto cuctx = ctx->CUDACtx();
  auto pre = p_cache->Pre(ctx);
  thrust::fill_n(cuctx->CTP(), pre.data(), pre.size(), 0.0);

  std::size_t bytes;
  hipcub::DeviceSegmentedReduce::Sum(nullptr, bytes, it, pre.data(), p_cache->Groups(), d_gptr.data(),
                                  d_gptr.data() + 1, cuctx->Stream());
  dh::TemporaryArray<char> temp(bytes);
  hipcub::DeviceSegmentedReduce::Sum(temp.data().get(), bytes, it, pre.data(), p_cache->Groups(),
                                  d_gptr.data(), d_gptr.data() + 1, cuctx->Stream());

  auto w_it =
      dh::MakeTransformIterator<double>(thrust::make_counting_iterator(0ul),
                                        [=] XGBOOST_DEVICE(std::size_t g) { return d_weight[g]; });
  auto n_weights = p_cache->Groups();
  auto sw = dh::Reduce(cuctx->CTP(), w_it, w_it + n_weights, 0.0, thrust::plus<double>{});
  auto sum =
      dh::Reduce(cuctx->CTP(), dh::tcbegin(pre), dh::tcend(pre), 0.0, thrust::plus<double>{});
  auto result = PackedReduceResult{sum, sw};
  return result;
}

PackedReduceResult NDCGScore(Context const *ctx, MetaInfo const &info,
                             HostDeviceVector<float> const &predt, bool minus,
                             std::shared_ptr<ltr::NDCGCache> p_cache) {
  CHECK(p_cache);

  auto const &p = p_cache->Param();
  auto d_weight = common::MakeOptionalWeights(ctx, info.weights_);
  if (!d_weight.Empty()) {
    CHECK_EQ(d_weight.weights.size(), p_cache->Groups());
  }
  auto d_label = info.labels.View(ctx->gpu_id).Slice(linalg::All(), 0);
  predt.SetDevice(ctx->gpu_id);
  auto d_predt = linalg::MakeTensorView(ctx, predt.ConstDeviceSpan(), predt.Size());

  auto d_group_ptr = p_cache->DataGroupPtr(ctx);

  auto d_inv_idcg = p_cache->InvIDCG(ctx);
  auto d_sorted_idx = p_cache->SortedIdx(ctx, d_predt.Values());
  auto d_out_dcg = p_cache->Dcg(ctx);

  ltr::cuda_impl::CalcQueriesDCG(ctx, d_label, d_sorted_idx, p.ndcg_exp_gain, d_group_ptr, p.TopK(),
                                 d_out_dcg);

  auto it = dh::MakeTransformIterator<PackedReduceResult>(
      thrust::make_counting_iterator(0ul), [=] XGBOOST_DEVICE(std::size_t i) {
        if (d_inv_idcg(i) <= 0.0) {
          return PackedReduceResult{minus ? 0.0 : 1.0, static_cast<double>(d_weight[i])};
        }
        return PackedReduceResult{d_out_dcg(i) * d_inv_idcg(i) * d_weight[i],
                                  static_cast<double>(d_weight[i])};
      });
  auto pair = thrust::reduce(ctx->CUDACtx()->CTP(), it, it + d_out_dcg.Size(),
                             PackedReduceResult{0.0, 0.0});
  return pair;
}

PackedReduceResult MAPScore(Context const *ctx, MetaInfo const &info,
                            HostDeviceVector<float> const &predt, bool minus,
                            std::shared_ptr<ltr::MAPCache> p_cache) {
  auto d_group_ptr = p_cache->DataGroupPtr(ctx);
  auto d_label = info.labels.View(ctx->gpu_id).Slice(linalg::All(), 0);

  predt.SetDevice(ctx->gpu_id);
  auto d_rank_idx = p_cache->SortedIdx(ctx, predt.ConstDeviceSpan());
  auto key_it = dh::MakeTransformIterator<std::size_t>(
      thrust::make_counting_iterator(0ul),
      [=] XGBOOST_DEVICE(std::size_t i) { return dh::SegmentId(d_group_ptr, i); });

  auto get_label = [=] XGBOOST_DEVICE(std::size_t i) {
    auto g = key_it[i];
    auto g_begin = d_group_ptr[g];
    auto g_end = d_group_ptr[g + 1];
    i -= g_begin;
    auto g_label = d_label.Slice(linalg::Range(g_begin, g_end));
    auto g_rank = d_rank_idx.subspan(g_begin, g_end - g_begin);
    return g_label(g_rank[i]);
  };
  auto it = dh::MakeTransformIterator<double>(thrust::make_counting_iterator(0ul), get_label);

  auto cuctx = ctx->CUDACtx();
  auto n_rel = p_cache->NumRelevant(ctx);
  thrust::inclusive_scan_by_key(cuctx->CTP(), key_it, key_it + d_label.Size(), it, n_rel.data());

  double topk = p_cache->Param().TopK();
  auto map = p_cache->Map(ctx);
  thrust::fill_n(cuctx->CTP(), map.data(), map.size(), 0.0);
  {
    auto val_it = dh::MakeTransformIterator<double>(
        thrust::make_counting_iterator(0ul), [=] XGBOOST_DEVICE(std::size_t i) {
          auto g = key_it[i];
          auto g_begin = d_group_ptr[g];
          auto g_end = d_group_ptr[g + 1];
          i -= g_begin;
          if (i >= topk) {
            return 0.0;
          }

          auto g_label = d_label.Slice(linalg::Range(g_begin, g_end));
          auto g_rank = d_rank_idx.subspan(g_begin, g_end - g_begin);
          auto label = g_label(g_rank[i]);

          auto g_n_rel = n_rel.subspan(g_begin, g_end - g_begin);
          auto nhits = g_n_rel[i];
          return nhits / static_cast<double>(i + 1) * label;
        });

    std::size_t bytes;
    hipcub::DeviceSegmentedReduce::Sum(nullptr, bytes, val_it, map.data(), p_cache->Groups(),
                                    d_group_ptr.data(), d_group_ptr.data() + 1, cuctx->Stream());
    dh::TemporaryArray<char> temp(bytes);
    hipcub::DeviceSegmentedReduce::Sum(temp.data().get(), bytes, val_it, map.data(), p_cache->Groups(),
                                    d_group_ptr.data(), d_group_ptr.data() + 1, cuctx->Stream());
  }

  PackedReduceResult result{0.0, 0.0};
  {
    auto d_weight = common::MakeOptionalWeights(ctx, info.weights_);
    if (!d_weight.Empty()) {
      CHECK_EQ(d_weight.weights.size(), p_cache->Groups());
    }
    auto val_it = dh::MakeTransformIterator<PackedReduceResult>(
        thrust::make_counting_iterator(0ul), [=] XGBOOST_DEVICE(std::size_t g) {
          auto g_begin = d_group_ptr[g];
          auto g_end = d_group_ptr[g + 1];
          auto g_n_rel = n_rel.subspan(g_begin, g_end - g_begin);
          if (!g_n_rel.empty() && g_n_rel.back() > 0.0) {
            return PackedReduceResult{map[g] * d_weight[g] / std::min(g_n_rel.back(), topk),
                                      static_cast<double>(d_weight[g])};
          }
          return PackedReduceResult{minus ? 0.0 : 1.0, static_cast<double>(d_weight[g])};
        });
    result =
        thrust::reduce(cuctx->CTP(), val_it, val_it + map.size(), PackedReduceResult{0.0, 0.0});
  }
  return result;
}
}  // namespace cuda_impl
}  // namespace xgboost::metric
