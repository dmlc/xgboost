#include "hip/hip_runtime.h"
/*!
 * Copyright 2020 by Contributors
 * \file rank_metric.cc
 * \brief prediction rank based metrics.
 * \author Kailong Chen, Tianqi Chen
 */
#include <rabit/rabit.h>
#include <dmlc/registry.h>

#include <xgboost/metric.h>
#include <xgboost/host_device_vector.h>
#include <thrust/iterator/discard_iterator.h>

#include <cmath>
#include <array>
#include <vector>

#include "metric_common.h"

#include "../common/math.h"
#include "../common/device_helpers.cuh"

namespace xgboost {
namespace metric {
// tag the this file, used by force static link later.
DMLC_REGISTRY_FILE_TAG(rank_metric_gpu);

/*! \brief Evaluate rank list on GPU */
template <typename EvalMetricT>
struct EvalRankGpu : public Metric, public EvalRankConfig {
 public:
  bst_float Eval(const HostDeviceVector<bst_float> &preds,
                 const MetaInfo &info,
                 bool distributed) override {
    // Sanity check is done by the caller
    std::vector<unsigned> tgptr(2, 0);
    tgptr[1] = static_cast<unsigned>(preds.Size());
    const std::vector<unsigned> &gptr = info.group_ptr_.size() == 0 ? tgptr : info.group_ptr_;

    const auto ngroups = static_cast<bst_omp_uint>(gptr.size() - 1);

    auto device = tparam_->gpu_id;
    dh::safe_cuda(hipSetDevice(device));

    info.labels_.SetDevice(device);
    preds.SetDevice(device);

    auto dpreds = preds.ConstDevicePointer();
    auto dlabels = info.labels_.ConstDevicePointer();

    // Sort all the predictions
    dh::SegmentSorter<float> segment_pred_sorter;
    segment_pred_sorter.SortItems(dpreds, preds.Size(), gptr);

    // Compute individual group metric and sum them up
    return EvalMetricT::EvalMetric(segment_pred_sorter, dlabels, *this);
  }

  const char* Name() const override {
    return name.c_str();
  }

  explicit EvalRankGpu(const char* name, const char* param) {
    using namespace std;  // NOLINT(*)
    if (param != nullptr) {
      std::ostringstream os;
      if (sscanf(param, "%u[-]?", &this->topn) == 1) {
        os << name << '@' << param;
        this->name = os.str();
      } else {
        os << name << param;
        this->name = os.str();
      }
      if (param[strlen(param) - 1] == '-') {
        this->minus = true;
      }
    } else {
      this->name = name;
    }
  }
};

/*! \brief Precision at N, for both classification and rank */
struct EvalPrecisionGpu {
 public:
  static double EvalMetric(const dh::SegmentSorter<float> &pred_sorter,
                           const float *dlabels,
                           const EvalRankConfig &ecfg) {
    // Group info on device
    const auto &dgroups = pred_sorter.GetGroupsSpan();
    const auto ngroups = pred_sorter.GetNumGroups();
    const auto &dgroup_idx = pred_sorter.GetGroupSegmentsSpan();

    // Original positions of the predictions after they have been sorted
    const auto &dpreds_orig_pos = pred_sorter.GetOriginalPositionsSpan();

    // First, determine non zero labels in the dataset individually
    auto DetermineNonTrivialLabelLambda = [=] __device__(uint32_t idx) {
      return (static_cast<unsigned>(dlabels[dpreds_orig_pos[idx]]) != 0) ? 1 : 0;
    };  // NOLINT

    // Find each group's metric sum
    dh::caching_device_vector<uint32_t> hits(ngroups, 0);
    const auto nitems = pred_sorter.GetNumItems();
    auto *dhits = hits.data().get();

    int device_id = -1;
    dh::safe_cuda(hipGetDevice(&device_id));
    // For each group item compute the aggregated precision
    dh::LaunchN(device_id, nitems, nullptr, [=] __device__(uint32_t idx) {
      const auto group_idx = dgroup_idx[idx];
      const auto group_begin = dgroups[group_idx];
      const auto ridx = idx - group_begin;
      if (ridx < ecfg.topn && DetermineNonTrivialLabelLambda(idx)) {
        atomicAdd(&dhits[group_idx], 1);
      }
    });

    // Allocator to be used for managing space overhead while performing reductions
    dh::XGBCachingDeviceAllocator<char> alloc;
    return static_cast<double>(thrust::reduce(thrust::cuda::par(alloc),
                                              hits.begin(), hits.end())) / ecfg.topn;
  }
};

/*! \brief NDCG: Normalized Discounted Cumulative Gain at N */
struct EvalNDCGGpu {
 public:
  static void ComputeDCG(const dh::SegmentSorter<float> &pred_sorter,
                         const float *dlabels,
                         const EvalRankConfig &ecfg,
                         // The order in which labels have to be accessed. The order is determined
                         // by sorting the predictions or the labels for the entire dataset
                         const xgboost::common::Span<const uint32_t> &dlabels_sort_order,
                         dh::caching_device_vector<double> *dcgptr) {
    dh::caching_device_vector<double> &dcgs(*dcgptr);
    // Group info on device
    const auto &dgroups = pred_sorter.GetGroupsSpan();
    const auto &dgroup_idx = pred_sorter.GetGroupSegmentsSpan();

    // First, determine non zero labels in the dataset individually
    auto DetermineNonTrivialLabelLambda = [=] __device__(uint32_t idx) {
      return (static_cast<unsigned>(dlabels[dlabels_sort_order[idx]]));
    };  // NOLINT

    // Find each group's DCG value
    const auto nitems = pred_sorter.GetNumItems();
    auto *ddcgs = dcgs.data().get();

    int device_id = -1;
    dh::safe_cuda(hipGetDevice(&device_id));

    // For each group item compute the aggregated precision
    dh::LaunchN(device_id, nitems, nullptr, [=] __device__(uint32_t idx) {
      const auto group_idx = dgroup_idx[idx];
      const auto group_begin = dgroups[group_idx];
      const auto ridx = idx - group_begin;
      auto label = DetermineNonTrivialLabelLambda(idx);
      if (ridx < ecfg.topn && label) {
        atomicAdd(&ddcgs[group_idx], ((1 << label) - 1) / std::log2(ridx + 2.0));
      }
    });
  }

  static double EvalMetric(const dh::SegmentSorter<float> &pred_sorter,
                           const float *dlabels,
                           const EvalRankConfig &ecfg) {
    // Sort the labels and compute IDCG
    dh::SegmentSorter<float> segment_label_sorter;
    segment_label_sorter.SortItems(dlabels, pred_sorter.GetNumItems(),
                                   pred_sorter.GetGroupSegmentsSpan());

    uint32_t ngroups = pred_sorter.GetNumGroups();

    dh::caching_device_vector<double> idcg(ngroups, 0);
    ComputeDCG(pred_sorter, dlabels, ecfg, segment_label_sorter.GetOriginalPositionsSpan(), &idcg);

    // Compute the DCG values next
    dh::caching_device_vector<double> dcg(ngroups, 0);
    ComputeDCG(pred_sorter, dlabels, ecfg, pred_sorter.GetOriginalPositionsSpan(), &dcg);

    double *ddcg = dcg.data().get();
    double *didcg = idcg.data().get();

    int device_id = -1;
    dh::safe_cuda(hipGetDevice(&device_id));
    // Compute the group's DCG and reduce it across all groups
    dh::LaunchN(device_id, ngroups, nullptr, [=] __device__(uint32_t gidx) {
      if (didcg[gidx] == 0.0f) {
        ddcg[gidx] = (ecfg.minus) ? 0.0f : 1.0f;
      } else {
        ddcg[gidx] /= didcg[gidx];
      }
    });

    // Allocator to be used for managing space overhead while performing reductions
    dh::XGBCachingDeviceAllocator<char> alloc;
    return thrust::reduce(thrust::cuda::par(alloc), dcg.begin(), dcg.end());
  }
};

/*! \brief Mean Average Precision at N, for both classification and rank */
struct EvalMAPGpu {
 public:
  static double EvalMetric(const dh::SegmentSorter<float> &pred_sorter,
                           const float *dlabels,
                           const EvalRankConfig &ecfg) {
    // Group info on device
    const auto &dgroups = pred_sorter.GetGroupsSpan();
    const auto ngroups = pred_sorter.GetNumGroups();
    const auto &dgroup_idx = pred_sorter.GetGroupSegmentsSpan();

    // Original positions of the predictions after they have been sorted
    const auto &dpreds_orig_pos = pred_sorter.GetOriginalPositionsSpan();

    // First, determine non zero labels in the dataset individually
    const auto nitems = pred_sorter.GetNumItems();
    dh::caching_device_vector<uint32_t> hits(nitems, 0);
    auto DetermineNonTrivialLabelLambda = [=] __device__(uint32_t idx) {
      return (static_cast<unsigned>(dlabels[dpreds_orig_pos[idx]]) != 0) ? 1 : 0;
    };  // NOLINT

    thrust::transform(thrust::make_counting_iterator(static_cast<uint32_t>(0)),
                      thrust::make_counting_iterator(nitems),
                      hits.begin(),
                      DetermineNonTrivialLabelLambda);

    // Allocator to be used by sort for managing space overhead while performing prefix scans
    dh::XGBCachingDeviceAllocator<char> alloc;

    // Next, prefix scan the nontrivial labels that are segmented to accumulate them.
    // This is required for computing the metric sum
    // Data segmented into different groups...
    thrust::inclusive_scan_by_key(thrust::cuda::par(alloc),
                                  dh::tcbegin(dgroup_idx), dh::tcend(dgroup_idx),
                                  hits.begin(),  // Input value
                                  hits.begin());  // In-place scan

    // Find each group's metric sum
    dh::caching_device_vector<double> sumap(ngroups, 0);
    auto *dsumap = sumap.data().get();
    const auto *dhits = hits.data().get();

    int device_id = -1;
    dh::safe_cuda(hipGetDevice(&device_id));
    // For each group item compute the aggregated precision
    dh::LaunchN(device_id, nitems, nullptr, [=] __device__(uint32_t idx) {
      if (DetermineNonTrivialLabelLambda(idx)) {
        const auto group_idx = dgroup_idx[idx];
        const auto group_begin = dgroups[group_idx];
        const auto ridx = idx - group_begin;
        if (ridx < ecfg.topn) {
          atomicAdd(&dsumap[group_idx],
                    static_cast<double>(dhits[idx]) / (ridx + 1));
        }
      }
    });

    // Aggregate the group's item precisions
    dh::LaunchN(device_id, ngroups, nullptr, [=] __device__(uint32_t gidx) {
      auto nhits = dgroups[gidx + 1] ? dhits[dgroups[gidx + 1] - 1] : 0;
      if (nhits != 0) {
        dsumap[gidx] /= nhits;
      } else {
        if (ecfg.minus) {
          dsumap[gidx] = 0;
        } else {
          dsumap[gidx] = 1;
        }
      }
    });

    return thrust::reduce(thrust::cuda::par(alloc), sumap.begin(), sumap.end());
  }
};

/*! \brief Area Under PR Curve metric computation for ranking datasets */
struct EvalAucPRGpu : public Metric {
 public:
  // This function object computes the item's positive/negative precision value
  class ComputeItemPrecision : public thrust::unary_function<uint32_t, float> {
   public:
    // The precision type to be computed
    enum class PrecisionType {
      kPositive,
      kNegative
    };

    XGBOOST_DEVICE ComputeItemPrecision(PrecisionType ptype,
                                        uint32_t ngroups,
                                        const float *dweights,
                                        const xgboost::common::Span<const uint32_t> &dgidxs,
                                        const float *dlabels)
      : ptype_(ptype), ngroups_(ngroups), dweights_(dweights), dgidxs_(dgidxs), dlabels_(dlabels) {}

    // Compute precision value for the prediction that was originally at 'idx'
    __device__ __forceinline__ float operator()(uint32_t idx) const {
      // For ranking task, weights are per-group
      // For binary classification task, weights are per-instance
      const auto wt = dweights_ == nullptr ? 1.0f : dweights_[ngroups_ == 1 ? idx : dgidxs_[idx]];
      return wt * (ptype_ == PrecisionType::kPositive ? dlabels_[idx] : (1.0f - dlabels_[idx]));
    }

   private:
    PrecisionType ptype_;  // Precision type to be computed
    uint32_t ngroups_;  // Number of groups in the dataset
    const float *dweights_;  // Instance/group weights
    const xgboost::common::Span<const uint32_t> dgidxs_;  // The group a given instance belongs to
    const float *dlabels_;  // Unsorted labels in the dataset
  };

  bst_float Eval(const HostDeviceVector<bst_float> &preds,
                 const MetaInfo &info,
                 bool distributed) override {
    // Sanity check is done by the caller
    std::vector<unsigned> tgptr(2, 0);
    tgptr[1] = static_cast<unsigned>(info.labels_.Size());
    const std::vector<unsigned> &gptr = info.group_ptr_.empty() ? tgptr : info.group_ptr_;

    auto device = tparam_->gpu_id;
    dh::safe_cuda(hipSetDevice(device));

    info.labels_.SetDevice(device);
    preds.SetDevice(device);
    info.weights_.SetDevice(device);

    auto dpreds = preds.ConstDevicePointer();
    auto dlabels = info.labels_.ConstDevicePointer();
    auto dweights = info.weights_.ConstDevicePointer();

    // Sort all the predictions
    dh::SegmentSorter<float> segment_pred_sorter;
    segment_pred_sorter.SortItems(dpreds, preds.Size(), gptr);

    const auto &dsorted_preds = segment_pred_sorter.GetItemsSpan();
    // Original positions of the predictions after they have been sorted
    const auto &dpreds_orig_pos = segment_pred_sorter.GetOriginalPositionsSpan();

    // Group info on device
    const auto &dgroups = segment_pred_sorter.GetGroupsSpan();
    uint32_t ngroups = segment_pred_sorter.GetNumGroups();
    const auto &dgroup_idx = segment_pred_sorter.GetGroupSegmentsSpan();

    // First, aggregate the positive and negative precision for each group
    dh::caching_device_vector<double> total_pos(ngroups, 0);
    dh::caching_device_vector<double> total_neg(ngroups, 0);

    // Allocator to be used for managing space overhead while performing transformed reductions
    dh::XGBCachingDeviceAllocator<char> alloc;

    // Compute each elements positive precision value and reduce them across groups concurrently.
    ComputeItemPrecision pos_prec_functor(ComputeItemPrecision::PrecisionType::kPositive,
                                          ngroups, dweights, dgroup_idx, dlabels);
    auto end_range =
      thrust::reduce_by_key(thrust::cuda::par(alloc),
                            dh::tcbegin(dgroup_idx), dh::tcend(dgroup_idx),
                            thrust::make_transform_iterator(
                              // The indices need not be sequential within a group, as we care only
                              // about the sum of positive precision values within a group
                              dh::tcbegin(segment_pred_sorter.GetOriginalPositionsSpan()),
                              pos_prec_functor),
                            thrust::make_discard_iterator(),  // We don't care for the group indices
                            total_pos.begin());  // Sum of positive precision values in the group
    CHECK(end_range.second - total_pos.begin() == total_pos.size());

    // Compute each elements negative precision value and reduce them across groups concurrently.
    ComputeItemPrecision neg_prec_functor(ComputeItemPrecision::PrecisionType::kNegative,
                                          ngroups, dweights, dgroup_idx, dlabels);
    end_range =
      thrust::reduce_by_key(thrust::cuda::par(alloc),
                            dh::tcbegin(dgroup_idx), dh::tcend(dgroup_idx),
                            thrust::make_transform_iterator(
                              // The indices need not be sequential within a group, as we care only
                              // about the sum of negative precision values within a group
                              dh::tcbegin(segment_pred_sorter.GetOriginalPositionsSpan()),
                              neg_prec_functor),
                            thrust::make_discard_iterator(),  // We don't care for the group indices
                            total_neg.begin());  // Sum of negative precision values in the group
    CHECK(end_range.second - total_neg.begin() == total_neg.size());

    const auto *dtotal_pos = total_pos.data().get();
    const auto *dtotal_neg = total_neg.data().get();

    // AUC sum for each group
    dh::caching_device_vector<double> sum_auc(ngroups, 0);
    // AUC error across all groups
    dh::caching_device_vector<int> auc_error(1, 0);
    auto *dsum_auc = sum_auc.data().get();
    auto *dauc_error = auc_error.data().get();

    int device_id = -1;
    dh::safe_cuda(hipGetDevice(&device_id));
    // For each group item compute the aggregated precision
    dh::LaunchN<1, 32>(device_id, ngroups, nullptr, [=] __device__(uint32_t gidx) {
      // We need pos > 0 && neg > 0
      if (dtotal_pos[gidx] <= 0.0 || dtotal_neg[gidx] <= 0.0) {
        atomicAdd(dauc_error, 1);
      } else {
        auto gbegin = dgroups[gidx];
        auto gend = dgroups[gidx + 1];
        // Calculate AUC
        double tp = 0.0, prevtp = 0.0, fp = 0.0, prevfp = 0.0, h = 0.0, a = 0.0, b = 0.0;
        for (auto i = gbegin; i < gend; ++i) {
          const auto wt = dweights == nullptr ? 1.0f
                                              : dweights[ngroups == 1 ? dpreds_orig_pos[i] : gidx];
          tp += wt * dlabels[dpreds_orig_pos[i]];
          fp += wt * (1.0f - dlabels[dpreds_orig_pos[i]]);
          if ((i < gend - 1 && dsorted_preds[i] != dsorted_preds[i + 1]) || (i == gend - 1)) {
            if (tp == prevtp) {
              a = 1.0;
              b = 0.0;
            } else {
              h = (fp - prevfp) / (tp - prevtp);
              a = 1.0 + h;
              b = (prevfp - h * prevtp) / dtotal_pos[gidx];
            }
            if (0.0 != b) {
              dsum_auc[gidx] += (tp / dtotal_pos[gidx] - prevtp / dtotal_pos[gidx] -
                                 b / a * (std::log(a * tp / dtotal_pos[gidx] + b) -
                                          std::log(a * prevtp / dtotal_pos[gidx] + b))) / a;
            } else {
              dsum_auc[gidx] += (tp / dtotal_pos[gidx] - prevtp / dtotal_pos[gidx]) / a;
            }
            prevtp = tp;
            prevfp = fp;
          }
        }

        // Sanity check
        if (tp < 0 || prevtp < 0 || fp < 0 || prevfp < 0) {
          // Check if we have any metric error thus far
          auto current_auc_error = atomicAdd(dauc_error, 0);
          KERNEL_CHECK(!current_auc_error);
        }
      }
    });

    const auto hsum_auc = thrust::reduce(thrust::cuda::par(alloc), sum_auc.begin(), sum_auc.end());
    const auto hauc_error = auc_error.back();  // Copy it back to host

    // Report average AUC-PR across all groups
    // In distributed mode, workers which only contains pos or neg samples
    // will be ignored when aggregate AUC-PR.
    bst_float dat[2] = {0.0f, 0.0f};
    if (hauc_error < static_cast<int>(ngroups)) {
      dat[0] = static_cast<bst_float>(hsum_auc);
      dat[1] = static_cast<bst_float>(static_cast<int>(ngroups) - hauc_error);
    }
    if (distributed) {
      rabit::Allreduce<rabit::op::Sum>(dat, 2);
    }
    CHECK_GT(dat[1], 0.0f)
      << "AUC-PR: the dataset only contains pos or neg samples";
    CHECK_LE(dat[0], dat[1]) << "AUC-PR: AUC > 1.0";
    return dat[0] / dat[1];
  }

  const char* Name() const override {
    return "aucpr";
  }
};

XGBOOST_REGISTER_GPU_METRIC(AucPRGpu, "aucpr")
.describe("Area under PR curve for rank computed on GPU.")
.set_body([](const char* param) { return new EvalAucPRGpu(); });

XGBOOST_REGISTER_GPU_METRIC(PrecisionGpu, "pre")
.describe("precision@k for rank computed on GPU.")
.set_body([](const char* param) { return new EvalRankGpu<EvalPrecisionGpu>("pre", param); });

XGBOOST_REGISTER_GPU_METRIC(NDCGGpu, "ndcg")
.describe("ndcg@k for rank computed on GPU.")
.set_body([](const char* param) { return new EvalRankGpu<EvalNDCGGpu>("ndcg", param); });

XGBOOST_REGISTER_GPU_METRIC(MAPGpu, "map")
.describe("map@k for rank computed on GPU.")
.set_body([](const char* param) { return new EvalRankGpu<EvalMAPGpu>("map", param); });
}  // namespace metric
}  // namespace xgboost
