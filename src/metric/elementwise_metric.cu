#include "hip/hip_runtime.h"
/*!
 * Copyright 2015-2019 by Contributors
 * \file elementwise_metric.cc
 * \brief evaluation metrics for elementwise binary or regression.
 * \author Kailong Chen, Tianqi Chen
 *
 *  The expressions like wsum == 0 ? esum : esum / wsum is used to handle empty dataset.
 */
#include <rabit/rabit.h>
#include <xgboost/metric.h>
#include <dmlc/registry.h>
#include <cmath>

#include "metric_common.h"
#include "../common/math.h"
#include "../common/common.h"

#if defined(XGBOOST_USE_CUDA)
#include <thrust/execution_policy.h>  // thrust::cuda::par
#include <thrust/functional.h>        // thrust::plus<>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>

#include "../common/device_helpers.cuh"
#endif  // XGBOOST_USE_CUDA

namespace xgboost {
namespace metric {
// tag the this file, used by force static link later.
DMLC_REGISTRY_FILE_TAG(elementwise_metric);

template <typename EvalRow>
class ElementWiseMetricsReduction {
 public:
  explicit ElementWiseMetricsReduction(EvalRow policy) : policy_(std::move(policy)) {}

  PackedReduceResult CpuReduceMetrics(
      const HostDeviceVector<bst_float>& weights,
      const HostDeviceVector<bst_float>& labels,
      const HostDeviceVector<bst_float>& preds) const {
    size_t ndata = labels.Size();

    const auto& h_labels = labels.HostVector();
    const auto& h_weights = weights.HostVector();
    const auto& h_preds = preds.HostVector();

    bst_float residue_sum = 0;
    bst_float weights_sum = 0;

#pragma omp parallel for reduction(+: residue_sum, weights_sum) schedule(static)
    for (omp_ulong i = 0; i < ndata; ++i) {
      const bst_float wt = h_weights.size() > 0 ? h_weights[i] : 1.0f;
      residue_sum += policy_.EvalRow(h_labels[i], h_preds[i]) * wt;
      weights_sum += wt;
    }
    PackedReduceResult res { residue_sum, weights_sum };
    return res;
  }

#if defined(XGBOOST_USE_CUDA)

  PackedReduceResult DeviceReduceMetrics(
      const HostDeviceVector<bst_float>& weights,
      const HostDeviceVector<bst_float>& labels,
      const HostDeviceVector<bst_float>& preds) {
    size_t n_data = preds.Size();

    thrust::counting_iterator<size_t> begin(0);
    thrust::counting_iterator<size_t> end = begin + n_data;

    auto s_label = labels.DeviceSpan();
    auto s_preds = preds.DeviceSpan();
    auto s_weights = weights.DeviceSpan();

    bool const is_null_weight = weights.Size() == 0;

    auto d_policy = policy_;

    dh::XGBCachingDeviceAllocator<char> alloc;
    PackedReduceResult result = thrust::transform_reduce(
        thrust::cuda::par(alloc),
        begin, end,
        [=] XGBOOST_DEVICE(size_t idx) {
          bst_float weight = is_null_weight ? 1.0f : s_weights[idx];

          bst_float residue = d_policy.EvalRow(s_label[idx], s_preds[idx]);
          residue *= weight;
          return PackedReduceResult{ residue, weight };
        },
        PackedReduceResult(),
        thrust::plus<PackedReduceResult>());

    return result;
  }

#endif  // XGBOOST_USE_CUDA

  PackedReduceResult Reduce(
      const GenericParameter &tparam,
      int device,
      const HostDeviceVector<bst_float>& weights,
      const HostDeviceVector<bst_float>& labels,
      const HostDeviceVector<bst_float>& preds) {
    PackedReduceResult result;

    if (device < 0) {
      result = CpuReduceMetrics(weights, labels, preds);
    }
#if defined(XGBOOST_USE_CUDA)
    else {  // NOLINT
      device_ = device;
      preds.SetDevice(device_);
      labels.SetDevice(device_);
      weights.SetDevice(device_);

      dh::safe_cuda(hipSetDevice(device_));
      result = DeviceReduceMetrics(weights, labels, preds);
    }
#endif  // defined(XGBOOST_USE_CUDA)
    return result;
  }

 private:
  EvalRow policy_;
#if defined(XGBOOST_USE_CUDA)
  int device_{-1};
#endif  // defined(XGBOOST_USE_CUDA)
};

struct EvalRowRMSE {
  char const *Name() const {
    return "rmse";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    bst_float diff = label - pred;
    return diff * diff;
  }
  static bst_float GetFinal(bst_float esum, bst_float wsum) {
    return wsum == 0 ? std::sqrt(esum) : std::sqrt(esum / wsum);
  }
};

struct EvalRowRMSLE {
  char const* Name() const {
    return "rmsle";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    bst_float diff = std::log1p(label) - std::log1p(pred);
    return diff * diff;
  }
  static bst_float GetFinal(bst_float esum, bst_float wsum) {
    return wsum == 0 ? std::sqrt(esum) : std::sqrt(esum / wsum);
  }
};

struct EvalRowMAE {
  const char *Name() const {
    return "mae";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    return std::abs(label - pred);
  }
  static bst_float GetFinal(bst_float esum, bst_float wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

struct EvalRowMAPE {
  const char *Name() const {
    return "mape";
  }
  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    return std::abs((label - pred) / label);
  }
  static bst_float GetFinal(bst_float esum, bst_float wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

struct EvalRowLogLoss {
  const char *Name() const {
    return "logloss";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float y, bst_float py) const {
    const bst_float eps = 1e-16f;
    const bst_float pneg = 1.0f - py;
    if (py < eps) {
      return -y * std::log(eps) - (1.0f - y)  * std::log(1.0f - eps);
    } else if (pneg < eps) {
      return -y * std::log(1.0f - eps) - (1.0f - y)  * std::log(eps);
    } else {
      return -y * std::log(py) - (1.0f - y) * std::log(pneg);
    }
  }

  static bst_float GetFinal(bst_float esum, bst_float wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

struct EvalRowMPHE {
  char const *Name() const {
    return "mphe";
  }
  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    bst_float diff = label - pred;
    return std::sqrt( 1 + diff * diff) - 1;
  }
  static bst_float GetFinal(bst_float esum, bst_float wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

struct EvalError {
  explicit EvalError(const char* param) {
    if (param != nullptr) {
      CHECK_EQ(sscanf(param, "%f", &threshold_), 1)
          << "unable to parse the threshold value for the error metric";
      has_param_ = true;
    } else {
      threshold_ = 0.5f;
      has_param_ = false;
    }
  }
  const char *Name() const {
    static std::string name;
    if (has_param_) {
      std::ostringstream os;
      os << "error";
      if (threshold_ != 0.5f) os << '@' << threshold_;
      name = os.str();
      return name.c_str();
    } else {
      return "error";
    }
  }

  XGBOOST_DEVICE bst_float EvalRow(
      bst_float label, bst_float pred) const {
    // assume label is in [0,1]
    return pred > threshold_ ? 1.0f - label : label;
  }

  static bst_float GetFinal(bst_float esum, bst_float wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }

 private:
  bst_float threshold_;
  bool has_param_;
};

struct EvalPoissonNegLogLik {
  const char *Name() const {
    return "poisson-nloglik";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float y, bst_float py) const {
    const bst_float eps = 1e-16f;
    if (py < eps) py = eps;
    return common::LogGamma(y + 1.0f) + py - std::log(py) * y;
  }

  static bst_float GetFinal(bst_float esum, bst_float wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

struct EvalGammaDeviance {
  const char *Name() const {
    return "gamma-deviance";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    bst_float epsilon = 1.0e-9;
    bst_float tmp = label / (pred + epsilon);
    return tmp - std::log(tmp) - 1;
  }
  static bst_float GetFinal(bst_float esum, bst_float wsum) {
    return 2 * esum;
  }
};

struct EvalGammaNLogLik {
  static const char *Name() {
    return "gamma-nloglik";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float y, bst_float py) const {
    bst_float psi = 1.0;
    bst_float theta = -1. / py;
    bst_float a = psi;
    bst_float b = -std::log(-theta);
    bst_float c = 1. / psi * std::log(y/psi) - std::log(y) - common::LogGamma(1. / psi);
    return -((y * theta - b) / a + c);
  }
  static bst_float GetFinal(bst_float esum, bst_float wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

struct EvalTweedieNLogLik {
  explicit EvalTweedieNLogLik(const char* param) {
    CHECK(param != nullptr)
        << "tweedie-nloglik must be in format tweedie-nloglik@rho";
    rho_ = atof(param);
    CHECK(rho_ < 2 && rho_ >= 1)
        << "tweedie variance power must be in interval [1, 2)";
  }
  const char *Name() const {
    static std::string name;
    std::ostringstream os;
    os << "tweedie-nloglik@" << rho_;
    name = os.str();
    return name.c_str();
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float y, bst_float p) const {
    bst_float a = y * std::exp((1 - rho_) * std::log(p)) / (1 - rho_);
    bst_float b = std::exp((2 - rho_) * std::log(p)) / (2 - rho_);
    return -a + b;
  }
  static bst_float GetFinal(bst_float esum, bst_float wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }

 protected:
  bst_float rho_;
};
/*!
 * \brief base class of element-wise evaluation
 * \tparam Derived the name of subclass
 */
template<typename Policy>
struct EvalEWiseBase : public Metric {
  EvalEWiseBase() = default;
  explicit EvalEWiseBase(char const* policy_param) :
    policy_{policy_param}, reducer_{policy_} {}

  bst_float Eval(const HostDeviceVector<bst_float>& preds,
                 const MetaInfo& info,
                 bool distributed) override {
    CHECK_EQ(preds.Size(), info.labels_.Size())
        << "label and prediction size not match, "
        << "hint: use merror or mlogloss for multi-class classification";
    int device = tparam_->gpu_id;

    auto result =
        reducer_.Reduce(*tparam_, device, info.weights_, info.labels_, preds);

    double dat[2] { result.Residue(), result.Weights() };

    if (distributed) {
      rabit::Allreduce<rabit::op::Sum>(dat, 2);
    }
    return Policy::GetFinal(dat[0], dat[1]);
  }

  const char* Name() const override {
    return policy_.Name();
  }

 private:
  Policy policy_;
  ElementWiseMetricsReduction<Policy> reducer_{policy_};
};

XGBOOST_REGISTER_METRIC(RMSE, "rmse")
.describe("Rooted mean square error.")
.set_body([](const char* param) { return new EvalEWiseBase<EvalRowRMSE>(); });

XGBOOST_REGISTER_METRIC(RMSLE, "rmsle")
.describe("Rooted mean square log error.")
.set_body([](const char* param) { return new EvalEWiseBase<EvalRowRMSLE>(); });

XGBOOST_REGISTER_METRIC(MAE, "mae")
.describe("Mean absolute error.")
.set_body([](const char* param) { return new EvalEWiseBase<EvalRowMAE>(); });

XGBOOST_REGISTER_METRIC(MAPE, "mape")
    .describe("Mean absolute percentage error.")
    .set_body([](const char* param) { return new EvalEWiseBase<EvalRowMAPE>(); });

XGBOOST_REGISTER_METRIC(MPHE, "mphe")
.describe("Mean Pseudo Huber error.")
.set_body([](const char* param) { return new EvalEWiseBase<EvalRowMPHE>(); });

XGBOOST_REGISTER_METRIC(LogLoss, "logloss")
.describe("Negative loglikelihood for logistic regression.")
.set_body([](const char* param) { return new EvalEWiseBase<EvalRowLogLoss>(); });

XGBOOST_REGISTER_METRIC(PossionNegLoglik, "poisson-nloglik")
.describe("Negative loglikelihood for poisson regression.")
.set_body([](const char* param) { return new EvalEWiseBase<EvalPoissonNegLogLik>(); });

XGBOOST_REGISTER_METRIC(GammaDeviance, "gamma-deviance")
.describe("Residual deviance for gamma regression.")
.set_body([](const char* param) { return new EvalEWiseBase<EvalGammaDeviance>(); });

XGBOOST_REGISTER_METRIC(GammaNLogLik, "gamma-nloglik")
.describe("Negative log-likelihood for gamma regression.")
.set_body([](const char* param) { return new EvalEWiseBase<EvalGammaNLogLik>(); });

XGBOOST_REGISTER_METRIC(Error, "error")
.describe("Binary classification error.")
.set_body([](const char* param) { return new EvalEWiseBase<EvalError>(param); });

XGBOOST_REGISTER_METRIC(TweedieNLogLik, "tweedie-nloglik")
.describe("tweedie-nloglik@rho for tweedie regression.")
.set_body([](const char* param) {
  return new EvalEWiseBase<EvalTweedieNLogLik>(param);
});

}  // namespace metric
}  // namespace xgboost
