#include "hip/hip_runtime.h"
/*!
 * Copyright 2015-2022 by XGBoost Contributors
 * \file elementwise_metric.cc
 * \brief evaluation metrics for elementwise binary or regression.
 * \author Kailong Chen, Tianqi Chen
 *
 *  The expressions like wsum == 0 ? esum : esum / wsum is used to handle empty dataset.
 */
#include <dmlc/registry.h>
#include <rabit/rabit.h>
#include <xgboost/metric.h>

#include <cmath>

#include "../common/common.h"
#include "../common/math.h"
#include "../common/pseudo_huber.h"
#include "../common/threading_utils.h"
#include "metric_common.h"

#if defined(XGBOOST_USE_CUDA)
#include <thrust/execution_policy.h>  // thrust::cuda::par
#include <thrust/functional.h>        // thrust::plus<>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>

#include "../common/device_helpers.cuh"
#endif  // XGBOOST_USE_CUDA

namespace xgboost {
namespace metric {
// tag the this file, used by force static link later.
DMLC_REGISTRY_FILE_TAG(elementwise_metric);

namespace {
/**
 * \brief Reduce function for element wise metrics.
 *
 *   The loss function should handle all the computation for each sample, including
 *   applying the weights.  A tuple of {error_i, weight_i} is expected as return.
 */
template <typename Fn>
PackedReduceResult Reduce(GenericParameter const* ctx, MetaInfo const& info, Fn&& loss) {
  PackedReduceResult result;
  auto labels = info.labels.View(ctx->gpu_id);
  if (ctx->IsCPU()) {
    auto n_threads = ctx->Threads();
    std::vector<double> score_tloc(n_threads, 0.0);
    std::vector<double> weight_tloc(n_threads, 0.0);
    // We sum over losses over all samples and targets instead of performing this for each
    // target since the first one approach more accurate while the second approach is used
    // for approximation in distributed setting.  For rmse:
    // - sqrt(1/w(sum_t0 + sum_t1 + ... + sum_tm))       // multi-target
    // - sqrt(avg_t0) + sqrt(avg_t1) + ... sqrt(avg_tm)  // distributed
    common::ParallelFor(info.labels.Size(), ctx->Threads(), [&](size_t i) {
      auto t_idx = omp_get_thread_num();
      size_t sample_id;
      size_t target_id;
      std::tie(sample_id, target_id) = linalg::UnravelIndex(i, labels.Shape());

      float v, wt;
      std::tie(v, wt) = loss(i, sample_id, target_id);
      score_tloc[t_idx] += v;
      weight_tloc[t_idx] += wt;
    });
    double residue_sum = std::accumulate(score_tloc.cbegin(), score_tloc.cend(), 0.0);
    double weights_sum = std::accumulate(weight_tloc.cbegin(), weight_tloc.cend(), 0.0);
    result = PackedReduceResult{residue_sum, weights_sum};
  } else {
#if defined(XGBOOST_USE_CUDA)
    dh::XGBCachingDeviceAllocator<char> alloc;
    thrust::counting_iterator<size_t> begin(0);
    thrust::counting_iterator<size_t> end = begin + labels.Size();
    result = thrust::transform_reduce(
        thrust::cuda::par(alloc), begin, end,
        [=] XGBOOST_DEVICE(size_t i) {
          auto idx = linalg::UnravelIndex(i, labels.Shape());
          auto sample_id = std::get<0>(idx);
          auto target_id = std::get<1>(idx);
          auto res = loss(i, sample_id, target_id);
          float v{std::get<0>(res)}, wt{std::get<1>(res)};
          return PackedReduceResult{v, wt};
        },
        PackedReduceResult{}, thrust::plus<PackedReduceResult>());
#else
    common::AssertGPUSupport();
#endif  //  defined(XGBOOST_USE_CUDA)
  }
  return result;
}
}  // anonymous namespace

struct EvalRowRMSE {
  char const *Name() const {
    return "rmse";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    bst_float diff = label - pred;
    return diff * diff;
  }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? std::sqrt(esum) : std::sqrt(esum / wsum);
  }
};

struct EvalRowRMSLE {
  char const* Name() const {
    return "rmsle";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    bst_float diff = std::log1p(label) - std::log1p(pred);
    return diff * diff;
  }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? std::sqrt(esum) : std::sqrt(esum / wsum);
  }
};

struct EvalRowMAE {
  const char *Name() const {
    return "mae";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    return std::abs(label - pred);
  }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

struct EvalRowMAPE {
  const char *Name() const {
    return "mape";
  }
  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    return std::abs((label - pred) / label);
  }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

namespace {
XGBOOST_DEVICE inline float LogLoss(float y, float py) {
  auto xlogy = [](float x, float y) {
    float eps = 1e-16;
    return (x - 0.0f == 0.0f) ? 0.0f : (x * std::log(std::max(y, eps)));
  };
  const bst_float pneg = 1.0f - py;
  return xlogy(-y, py) + xlogy(-(1.0f - y), pneg);
}
}  // anonymous namespace

struct EvalRowLogLoss {
  const char *Name() const {
    return "logloss";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float y, bst_float py) const { return LogLoss(y, py); }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

class PseudoErrorLoss : public Metric {
  PesudoHuberParam param_;

 public:
  const char* Name() const override { return "mphe"; }
  void Configure(Args const& args) override { param_.UpdateAllowUnknown(args); }
  void LoadConfig(Json const& in) override { FromJson(in["pseudo_huber_param"], &param_); }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String(this->Name());
    out["pseudo_huber_param"] = ToJson(param_);
  }

  double Eval(const HostDeviceVector<bst_float>& preds, const MetaInfo& info) override {
    CHECK_EQ(info.labels.Shape(0), info.num_row_);
    auto labels = info.labels.View(tparam_->gpu_id);
    preds.SetDevice(tparam_->gpu_id);
    auto predts = tparam_->IsCPU() ? preds.ConstHostSpan() : preds.ConstDeviceSpan();
    info.weights_.SetDevice(tparam_->gpu_id);
    common::OptionalWeights weights(tparam_->IsCPU() ? info.weights_.ConstHostSpan()
                                                     : info.weights_.ConstDeviceSpan());
    float slope = this->param_.huber_slope;
    CHECK_NE(slope, 0.0) << "slope for pseudo huber cannot be 0.";
    PackedReduceResult result =
        Reduce(tparam_, info, [=] XGBOOST_DEVICE(size_t i, size_t sample_id, size_t target_id) {
          float wt = weights[sample_id];
          auto a = labels(sample_id, target_id) - predts[i];
          auto v = common::Sqr(slope) * (std::sqrt((1 + common::Sqr(a / slope))) - 1) * wt;
          return std::make_tuple(v, wt);
        });
    double dat[2]{result.Residue(), result.Weights()};
    if (rabit::IsDistributed()) {
      rabit::Allreduce<rabit::op::Sum>(dat, 2);
    }
    return EvalRowMAPE::GetFinal(dat[0], dat[1]);
  }
};

struct EvalError {
  explicit EvalError(const char* param) {
    if (param != nullptr) {
      CHECK_EQ(sscanf(param, "%f", &threshold_), 1)
          << "unable to parse the threshold value for the error metric";
      has_param_ = true;
    } else {
      threshold_ = 0.5f;
      has_param_ = false;
    }
  }
  const char *Name() const {
    static std::string name;
    if (has_param_) {
      std::ostringstream os;
      os << "error";
      if (threshold_ != 0.5f) os << '@' << threshold_;
      name = os.str();
      return name.c_str();
    } else {
      return "error";
    }
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    // assume label is in [0,1]
    return pred > threshold_ ? 1.0f - label : label;
  }

  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }

 private:
  bst_float threshold_;
  bool has_param_;
};

struct EvalPoissonNegLogLik {
  const char *Name() const {
    return "poisson-nloglik";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float y, bst_float py) const {
    const bst_float eps = 1e-16f;
    if (py < eps) py = eps;
    return common::LogGamma(y + 1.0f) + py - std::log(py) * y;
  }

  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

/**
 * Gamma deviance
 *
 *   Expected input:
 *   label >= 0
 *   predt >= 0
 */
struct EvalGammaDeviance {
  const char *Name() const { return "gamma-deviance"; }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float predt) const {
    predt += kRtEps;
    label += kRtEps;
    return std::log(predt / label) + label / predt - 1;
  }

  static double GetFinal(double esum, double wsum) {
    if (wsum <= 0) {
      wsum = kRtEps;
    }
    return 2 * esum / wsum;
  }
};

struct EvalGammaNLogLik {
  static const char *Name() {
    return "gamma-nloglik";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float y, bst_float py) const {
    py = std::max(py, 1e-6f);
    // hardcoded dispersion.
    float constexpr kPsi = 1.0;
    bst_float theta = -1. / py;
    bst_float a = kPsi;
    float b = -std::log(-theta);
    // c = 1. / kPsi^2 * std::log(y/kPsi) - std::log(y) - common::LogGamma(1. / kPsi);
    //   = 1.0f        * std::log(y)      - std::log(y) - 0 = 0
    float c = 0;
    // general form for exponential family.
    return -((y * theta - b) / a + c);
  }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

struct EvalTweedieNLogLik {
  explicit EvalTweedieNLogLik(const char* param) {
    CHECK(param != nullptr)
        << "tweedie-nloglik must be in format tweedie-nloglik@rho";
    rho_ = atof(param);
    CHECK(rho_ < 2 && rho_ >= 1)
        << "tweedie variance power must be in interval [1, 2)";
  }
  const char *Name() const {
    static std::string name;
    std::ostringstream os;
    os << "tweedie-nloglik@" << rho_;
    name = os.str();
    return name.c_str();
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float y, bst_float p) const {
    bst_float a = y * std::exp((1 - rho_) * std::log(p)) / (1 - rho_);
    bst_float b = std::exp((2 - rho_) * std::log(p)) / (2 - rho_);
    return -a + b;
  }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }

 protected:
  bst_float rho_;
};
/*!
 * \brief base class of element-wise evaluation
 * \tparam Derived the name of subclass
 */
template <typename Policy>
struct EvalEWiseBase : public Metric {
  EvalEWiseBase() = default;
  explicit EvalEWiseBase(char const* policy_param) : policy_{policy_param} {}

  double Eval(HostDeviceVector<bst_float> const& preds, const MetaInfo& info) override {
    CHECK_EQ(preds.Size(), info.labels.Size())
        << "label and prediction size not match, "
        << "hint: use merror or mlogloss for multi-class classification";
    if (info.labels.Size() != 0) {
      CHECK_NE(info.labels.Shape(1), 0);
    }
    auto labels = info.labels.View(tparam_->gpu_id);
    info.weights_.SetDevice(tparam_->gpu_id);
    common::OptionalWeights weights(tparam_->IsCPU() ? info.weights_.ConstHostSpan()
                                                     : info.weights_.ConstDeviceSpan());
    preds.SetDevice(tparam_->gpu_id);
    auto predts = tparam_->IsCPU() ? preds.ConstHostSpan() : preds.ConstDeviceSpan();

    auto d_policy = policy_;
    auto result =
        Reduce(tparam_, info, [=] XGBOOST_DEVICE(size_t i, size_t sample_id, size_t target_id) {
          float wt = weights[sample_id];
          float residue = d_policy.EvalRow(labels(sample_id, target_id), predts[i]);
          residue *= wt;
          return std::make_tuple(residue, wt);
        });

    double dat[2]{result.Residue(), result.Weights()};
    rabit::Allreduce<rabit::op::Sum>(dat, 2);
    return Policy::GetFinal(dat[0], dat[1]);
  }

  const char* Name() const override { return policy_.Name(); }

 private:
  Policy policy_;
};

XGBOOST_REGISTER_METRIC(RMSE, "rmse")
    .describe("Rooted mean square error.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalRowRMSE>(); });

XGBOOST_REGISTER_METRIC(RMSLE, "rmsle")
    .describe("Rooted mean square log error.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalRowRMSLE>(); });

XGBOOST_REGISTER_METRIC(MAE, "mae").describe("Mean absolute error.").set_body([](const char*) {
  return new EvalEWiseBase<EvalRowMAE>();
});

XGBOOST_REGISTER_METRIC(MAPE, "mape")
    .describe("Mean absolute percentage error.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalRowMAPE>(); });

XGBOOST_REGISTER_METRIC(LogLoss, "logloss")
    .describe("Negative loglikelihood for logistic regression.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalRowLogLoss>(); });

XGBOOST_REGISTER_METRIC(PseudoErrorLoss, "mphe")
    .describe("Mean Pseudo-huber error.")
    .set_body([](const char*) { return new PseudoErrorLoss{}; });

XGBOOST_REGISTER_METRIC(PossionNegLoglik, "poisson-nloglik")
    .describe("Negative loglikelihood for poisson regression.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalPoissonNegLogLik>(); });

XGBOOST_REGISTER_METRIC(GammaDeviance, "gamma-deviance")
    .describe("Residual deviance for gamma regression.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalGammaDeviance>(); });

XGBOOST_REGISTER_METRIC(GammaNLogLik, "gamma-nloglik")
    .describe("Negative log-likelihood for gamma regression.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalGammaNLogLik>(); });

XGBOOST_REGISTER_METRIC(Error, "error")
.describe("Binary classification error.")
.set_body([](const char* param) { return new EvalEWiseBase<EvalError>(param); });

XGBOOST_REGISTER_METRIC(TweedieNLogLik, "tweedie-nloglik")
.describe("tweedie-nloglik@rho for tweedie regression.")
.set_body([](const char* param) {
  return new EvalEWiseBase<EvalTweedieNLogLik>(param);
});
}  // namespace metric
}  // namespace xgboost
