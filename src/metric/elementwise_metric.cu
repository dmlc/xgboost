#include "hip/hip_runtime.h"
/**
 * Copyright 2015-2023 by XGBoost Contributors
 * \file elementwise_metric.cu
 * \brief evaluation metrics for elementwise binary or regression.
 * \author Kailong Chen, Tianqi Chen
 *
 *  The expressions like wsum == 0 ? esum : esum / wsum is used to handle empty dataset.
 */
#include <dmlc/registry.h>

#include <array>
#include <cmath>

#include "../collective/communicator-inl.h"
#include "../common/common.h"           // MetricNoCache
#include "../common/math.h"
#include "../common/optional_weight.h"  // OptionalWeights
#include "../common/pseudo_huber.h"
#include "../common/quantile_loss_utils.h"  // QuantileLossParam
#include "../common/threading_utils.h"
#include "metric_common.h"
#include "xgboost/metric.h"

#if defined(XGBOOST_USE_CUDA)
#include <thrust/execution_policy.h>  // thrust::cuda::par
#include <thrust/functional.h>        // thrust::plus<>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform_reduce.h>

#include "../common/device_helpers.cuh"
#endif  // XGBOOST_USE_CUDA

namespace xgboost {
namespace metric {
// tag the this file, used by force static link later.
DMLC_REGISTRY_FILE_TAG(elementwise_metric);

namespace {
/**
 * \brief Reduce function for element wise metrics.
 *
 *   The loss function should handle all the computation for each sample, including
 *   applying the weights.  A tuple of {error_i, weight_i} is expected as return.
 */
template <typename Fn>
PackedReduceResult Reduce(Context const* ctx, MetaInfo const& info, Fn&& loss) {
  PackedReduceResult result;
  auto labels = info.labels.View(ctx->gpu_id);
  if (ctx->IsCPU()) {
    auto n_threads = ctx->Threads();
    std::vector<double> score_tloc(n_threads, 0.0);
    std::vector<double> weight_tloc(n_threads, 0.0);
    // We sum over losses over all samples and targets instead of performing this for each
    // target since the first one approach more accurate while the second approach is used
    // for approximation in distributed setting.  For rmse:
    // - sqrt(1/w(sum_t0 + sum_t1 + ... + sum_tm))       // multi-target
    // - sqrt(avg_t0) + sqrt(avg_t1) + ... sqrt(avg_tm)  // distributed
    common::ParallelFor(info.labels.Size(), ctx->Threads(), [&](size_t i) {
      auto t_idx = omp_get_thread_num();
      size_t sample_id;
      size_t target_id;
      std::tie(sample_id, target_id) = linalg::UnravelIndex(i, labels.Shape());

      float v, wt;
      std::tie(v, wt) = loss(i, sample_id, target_id);
      score_tloc[t_idx] += v;
      weight_tloc[t_idx] += wt;
    });
    double residue_sum = std::accumulate(score_tloc.cbegin(), score_tloc.cend(), 0.0);
    double weights_sum = std::accumulate(weight_tloc.cbegin(), weight_tloc.cend(), 0.0);
    result = PackedReduceResult{residue_sum, weights_sum};
  } else {
#if defined(XGBOOST_USE_CUDA)
    dh::XGBCachingDeviceAllocator<char> alloc;
    thrust::counting_iterator<size_t> begin(0);
    thrust::counting_iterator<size_t> end = begin + labels.Size();
    result = thrust::transform_reduce(
        thrust::cuda::par(alloc), begin, end,
        [=] XGBOOST_DEVICE(size_t i) {
          auto idx = linalg::UnravelIndex(i, labels.Shape());
          auto sample_id = std::get<0>(idx);
          auto target_id = std::get<1>(idx);
          auto res = loss(i, sample_id, target_id);
          float v{std::get<0>(res)}, wt{std::get<1>(res)};
          return PackedReduceResult{v, wt};
        },
        PackedReduceResult{}, thrust::plus<PackedReduceResult>());
#else
    common::AssertGPUSupport();
#endif  //  defined(XGBOOST_USE_CUDA)
  }
  return result;
}
}  // anonymous namespace

struct EvalRowRMSE {
  char const *Name() const {
    return "rmse";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    bst_float diff = label - pred;
    return diff * diff;
  }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? std::sqrt(esum) : std::sqrt(esum / wsum);
  }
};

struct EvalRowRMSLE {
  char const* Name() const {
    return "rmsle";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    bst_float diff = std::log1p(label) - std::log1p(pred);
    return diff * diff;
  }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? std::sqrt(esum) : std::sqrt(esum / wsum);
  }
};

struct EvalRowMAE {
  const char *Name() const {
    return "mae";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    return std::abs(label - pred);
  }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

struct EvalRowMAPE {
  const char *Name() const {
    return "mape";
  }
  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    return std::abs((label - pred) / label);
  }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

namespace {
XGBOOST_DEVICE inline float LogLoss(float y, float py) {
  auto xlogy = [](float x, float y) {
    float eps = 1e-16;
    return (x - 0.0f == 0.0f) ? 0.0f : (x * std::log(std::max(y, eps)));
  };
  const bst_float pneg = 1.0f - py;
  return xlogy(-y, py) + xlogy(-(1.0f - y), pneg);
}
}  // anonymous namespace

struct EvalRowLogLoss {
  const char *Name() const {
    return "logloss";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float y, bst_float py) const { return LogLoss(y, py); }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

class PseudoErrorLoss : public MetricNoCache {
  PesudoHuberParam param_;

 public:
  const char* Name() const override { return "mphe"; }
  void Configure(Args const& args) override { param_.UpdateAllowUnknown(args); }
  void LoadConfig(Json const& in) override { FromJson(in["pseudo_huber_param"], &param_); }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String(this->Name());
    out["pseudo_huber_param"] = ToJson(param_);
  }

  double Eval(const HostDeviceVector<bst_float>& preds, const MetaInfo& info) override {
    CHECK_EQ(info.labels.Shape(0), info.num_row_);
    auto labels = info.labels.View(ctx_->gpu_id);
    preds.SetDevice(ctx_->gpu_id);
    auto predts = ctx_->IsCPU() ? preds.ConstHostSpan() : preds.ConstDeviceSpan();
    info.weights_.SetDevice(ctx_->gpu_id);
    common::OptionalWeights weights(ctx_->IsCPU() ? info.weights_.ConstHostSpan()
                                                     : info.weights_.ConstDeviceSpan());
    float slope = this->param_.huber_slope;
    CHECK_NE(slope, 0.0) << "slope for pseudo huber cannot be 0.";
    PackedReduceResult result =
        Reduce(ctx_, info, [=] XGBOOST_DEVICE(size_t i, size_t sample_id, size_t target_id) {
          float wt = weights[sample_id];
          auto a = labels(sample_id, target_id) - predts[i];
          auto v = common::Sqr(slope) * (std::sqrt((1 + common::Sqr(a / slope))) - 1) * wt;
          return std::make_tuple(v, wt);
        });
    std::array<double, 2> dat{result.Residue(), result.Weights()};
    collective::GlobalSum(info, &dat);
    return EvalRowMAPE::GetFinal(dat[0], dat[1]);
  }
};

struct EvalError {
  explicit EvalError(const char* param) {
    if (param != nullptr) {
      CHECK_EQ(sscanf(param, "%f", &threshold_), 1)
          << "unable to parse the threshold value for the error metric";
      has_param_ = true;
    } else {
      threshold_ = 0.5f;
      has_param_ = false;
    }
  }
  const char *Name() const {
    static thread_local std::string name;
    if (has_param_) {
      std::ostringstream os;
      os << "error";
      if (threshold_ != 0.5f) os << '@' << threshold_;
      name = os.str();
      return name.c_str();
    } else {
      return "error";
    }
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float pred) const {
    // assume label is in [0,1]
    return pred > threshold_ ? 1.0f - label : label;
  }

  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }

 private:
  bst_float threshold_;
  bool has_param_;
};

struct EvalPoissonNegLogLik {
  const char *Name() const {
    return "poisson-nloglik";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float y, bst_float py) const {
    const bst_float eps = 1e-16f;
    if (py < eps) py = eps;
    return common::LogGamma(y + 1.0f) + py - std::log(py) * y;
  }

  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

/**
 * Gamma deviance
 *
 *   Expected input:
 *   label >= 0
 *   predt >= 0
 */
struct EvalGammaDeviance {
  const char *Name() const { return "gamma-deviance"; }

  XGBOOST_DEVICE bst_float EvalRow(bst_float label, bst_float predt) const {
    predt += kRtEps;
    label += kRtEps;
    return std::log(predt / label) + label / predt - 1;
  }

  static double GetFinal(double esum, double wsum) {
    if (wsum <= 0) {
      wsum = kRtEps;
    }
    return 2 * esum / wsum;
  }
};

struct EvalGammaNLogLik {
  static const char *Name() {
    return "gamma-nloglik";
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float y, bst_float py) const {
    py = std::max(py, 1e-6f);
    // hardcoded dispersion.
    float constexpr kPsi = 1.0;
    bst_float theta = -1. / py;
    bst_float a = kPsi;
    float b = -std::log(-theta);
    // c = 1. / kPsi^2 * std::log(y/kPsi) - std::log(y) - common::LogGamma(1. / kPsi);
    //   = 1.0f        * std::log(y)      - std::log(y) - 0 = 0
    float c = 0;
    // general form for exponential family.
    return -((y * theta - b) / a + c);
  }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }
};

struct EvalTweedieNLogLik {
  explicit EvalTweedieNLogLik(const char* param) {
    CHECK(param != nullptr)
        << "tweedie-nloglik must be in format tweedie-nloglik@rho";
    rho_ = atof(param);
    CHECK(rho_ < 2 && rho_ >= 1)
        << "tweedie variance power must be in interval [1, 2)";
  }
  const char *Name() const {
    static thread_local std::string name;
    std::ostringstream os;
    os << "tweedie-nloglik@" << rho_;
    name = os.str();
    return name.c_str();
  }

  XGBOOST_DEVICE bst_float EvalRow(bst_float y, bst_float p) const {
    bst_float a = y * std::exp((1 - rho_) * std::log(p)) / (1 - rho_);
    bst_float b = std::exp((2 - rho_) * std::log(p)) / (2 - rho_);
    return -a + b;
  }
  static double GetFinal(double esum, double wsum) {
    return wsum == 0 ? esum : esum / wsum;
  }

 protected:
  bst_float rho_;
};
/*!
 * \brief base class of element-wise evaluation
 * \tparam Derived the name of subclass
 */
template <typename Policy>
struct EvalEWiseBase : public MetricNoCache {
  EvalEWiseBase() = default;
  explicit EvalEWiseBase(char const* policy_param) : policy_{policy_param} {}

  double Eval(HostDeviceVector<bst_float> const& preds, const MetaInfo& info) override {
    CHECK_EQ(preds.Size(), info.labels.Size())
        << "label and prediction size not match, "
        << "hint: use merror or mlogloss for multi-class classification";
    if (info.labels.Size() != 0) {
      CHECK_NE(info.labels.Shape(1), 0);
    }
    auto labels = info.labels.View(ctx_->gpu_id);
    info.weights_.SetDevice(ctx_->gpu_id);
    common::OptionalWeights weights(ctx_->IsCPU() ? info.weights_.ConstHostSpan()
                                                     : info.weights_.ConstDeviceSpan());
    preds.SetDevice(ctx_->gpu_id);
    auto predts = ctx_->IsCPU() ? preds.ConstHostSpan() : preds.ConstDeviceSpan();

    auto d_policy = policy_;
    auto result =
        Reduce(ctx_, info, [=] XGBOOST_DEVICE(size_t i, size_t sample_id, size_t target_id) {
          float wt = weights[sample_id];
          float residue = d_policy.EvalRow(labels(sample_id, target_id), predts[i]);
          residue *= wt;
          return std::make_tuple(residue, wt);
        });

    std::array<double, 2> dat{result.Residue(), result.Weights()};
    collective::GlobalSum(info, &dat);
    return Policy::GetFinal(dat[0], dat[1]);
  }

  const char* Name() const override { return policy_.Name(); }

 private:
  Policy policy_;
};

XGBOOST_REGISTER_METRIC(RMSE, "rmse")
    .describe("Rooted mean square error.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalRowRMSE>(); });

XGBOOST_REGISTER_METRIC(RMSLE, "rmsle")
    .describe("Rooted mean square log error.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalRowRMSLE>(); });

XGBOOST_REGISTER_METRIC(MAE, "mae").describe("Mean absolute error.").set_body([](const char*) {
  return new EvalEWiseBase<EvalRowMAE>();
});

XGBOOST_REGISTER_METRIC(MAPE, "mape")
    .describe("Mean absolute percentage error.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalRowMAPE>(); });

XGBOOST_REGISTER_METRIC(LogLoss, "logloss")
    .describe("Negative loglikelihood for logistic regression.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalRowLogLoss>(); });

XGBOOST_REGISTER_METRIC(PseudoErrorLoss, "mphe")
    .describe("Mean Pseudo-huber error.")
    .set_body([](const char*) { return new PseudoErrorLoss{}; });

XGBOOST_REGISTER_METRIC(PossionNegLoglik, "poisson-nloglik")
    .describe("Negative loglikelihood for poisson regression.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalPoissonNegLogLik>(); });

XGBOOST_REGISTER_METRIC(GammaDeviance, "gamma-deviance")
    .describe("Residual deviance for gamma regression.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalGammaDeviance>(); });

XGBOOST_REGISTER_METRIC(GammaNLogLik, "gamma-nloglik")
    .describe("Negative log-likelihood for gamma regression.")
    .set_body([](const char*) { return new EvalEWiseBase<EvalGammaNLogLik>(); });

XGBOOST_REGISTER_METRIC(Error, "error")
.describe("Binary classification error.")
.set_body([](const char* param) { return new EvalEWiseBase<EvalError>(param); });

XGBOOST_REGISTER_METRIC(TweedieNLogLik, "tweedie-nloglik")
.describe("tweedie-nloglik@rho for tweedie regression.")
.set_body([](const char* param) {
  return new EvalEWiseBase<EvalTweedieNLogLik>(param);
});

class QuantileError : public MetricNoCache {
  HostDeviceVector<float> alpha_;
  common::QuantileLossParam param_;

 public:
  void Configure(Args const& args) override {
    param_.UpdateAllowUnknown(args);
    param_.Validate();
    alpha_.HostVector() = param_.quantile_alpha.Get();
  }

  double Eval(HostDeviceVector<bst_float> const& preds, const MetaInfo& info) override {
    CHECK(!alpha_.Empty());
    if (info.num_row_ == 0) {
      // empty DMatrix on distributed env
      std::array<double, 2> dat{0.0, 0.0};
      collective::GlobalSum(info, &dat);
      CHECK_GT(dat[1], 0);
      return dat[0] / dat[1];
    }

    auto const* ctx = ctx_;
    auto y_true = info.labels.View(ctx->gpu_id);
    preds.SetDevice(ctx->gpu_id);
    alpha_.SetDevice(ctx->gpu_id);
    auto alpha = ctx->IsCPU() ? alpha_.ConstHostSpan() : alpha_.ConstDeviceSpan();
    std::size_t n_targets = preds.Size() / info.num_row_ / alpha_.Size();
    CHECK_NE(n_targets, 0);
    auto y_predt = linalg::MakeTensorView(ctx, &preds, static_cast<std::size_t>(info.num_row_),
                                          alpha_.Size(), n_targets);

    info.weights_.SetDevice(ctx->gpu_id);
    common::OptionalWeights weight{ctx->IsCPU() ? info.weights_.ConstHostSpan()
                                                : info.weights_.ConstDeviceSpan()};

    auto result = Reduce(
        ctx, info, [=] XGBOOST_DEVICE(std::size_t i, std::size_t sample_id, std::size_t target_id) {
          auto idx = linalg::UnravelIndex(i, y_predt.Shape());
          sample_id = std::get<0>(idx);
          std::size_t quantile_id = std::get<1>(idx);
          target_id = std::get<2>(idx);

          auto loss = [a = alpha[quantile_id]](float p, float y) {
            auto d = y - p;
            float sign = d >= 0.0f;
            auto res = (a * sign * d) - (1.0f - a) * (1.0f - sign) * d;
            return res;
          };
          auto w = weight[sample_id];
          auto l =
              loss(y_predt(sample_id, quantile_id, target_id), y_true(sample_id, target_id)) * w;
          return std::make_tuple(l, w);
        });
    std::array<double, 2> dat{result.Residue(), result.Weights()};
    collective::GlobalSum(info, &dat);
    CHECK_GT(dat[1], 0);
    return dat[0] / dat[1];
  }

  const char* Name() const override { return "quantile"; }
  void LoadConfig(Json const& in) override {
    auto const& obj = get<Object const>(in);
    auto it = obj.find("quantile_loss_param");
    if (it != obj.cend()) {
      FromJson(it->second, &param_);
      auto const& name = get<String const>(in["name"]);
      CHECK_EQ(name, "quantile");
    }
  }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String(this->Name());
    out["quantile_loss_param"] = ToJson(param_);
  }
};

XGBOOST_REGISTER_METRIC(QuantileError, "quantile")
    .describe("Quantile regression error.")
    .set_body([](const char*) { return new QuantileError{}; });
}  // namespace metric
}  // namespace xgboost
