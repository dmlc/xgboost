#include "hip/hip_runtime.h"
/*!
 * Copyright 2021 by XGBoost Contributors
 */
#include <thrust/scan.h>
#include <hipcub/hipcub.hpp>
#include <cassert>
#include <limits>
#include <memory>
#include <utility>
#include <tuple>

#include "rabit/rabit.h"
#include "xgboost/span.h"
#include "xgboost/data.h"
#include "auc.h"
#include "../common/device_helpers.cuh"
#include "../common/ranking_utils.cuh"

namespace xgboost {
namespace metric {
namespace {
template <typename T>
using Discard = thrust::discard_iterator<T>;

struct GetWeightOp {
  common::Span<float const> weights;
  common::Span<size_t const> sorted_idx;

  __device__ float operator()(size_t i) const {
    return weights.empty() ? 1.0f : weights[sorted_idx[i]];
  }
};
}  // namespace

/**
 * A cache to GPU data to avoid reallocating memory.
 */
struct DeviceAUCCache {
  // Pair of FP/TP
  using Pair = thrust::pair<float, float>;
  // index sorted by prediction value
  dh::device_vector<size_t> sorted_idx;
  // track FP/TP for computation on trapesoid area
  dh::device_vector<Pair> fptp;
  // track FP_PREV/TP_PREV for computation on trapesoid area
  dh::device_vector<Pair> neg_pos;
  // index of unique prediction values.
  dh::device_vector<size_t> unique_idx;
  // p^T: transposed prediction matrix, used by MultiClassAUC
  dh::device_vector<float> predts_t;
  std::unique_ptr<dh::AllReducer> reducer;

  void Init(common::Span<float const> predts, bool is_multi, int32_t device) {
    if (sorted_idx.size() != predts.size()) {
      sorted_idx.resize(predts.size());
      fptp.resize(sorted_idx.size());
      unique_idx.resize(sorted_idx.size());
      neg_pos.resize(sorted_idx.size());
      if (is_multi) {
        predts_t.resize(sorted_idx.size());
      }
    }
    if (is_multi && !reducer) {
      reducer.reset(new dh::AllReducer);
      reducer->Init(device);
    }
  }
};

/**
 * The GPU implementation uses same calculation as CPU with a few more steps to distribute
 * work across threads:
 *
 * - Run scan to obtain TP/FP values, which are right coordinates of trapesoid.
 * - Find distinct prediction values and get the corresponding FP_PREV/TP_PREV value,
 *   which are left coordinates of trapesoids.
 * - Reduce the scan array into 1 AUC value.
 */
std::tuple<float, float, float>
GPUBinaryAUC(common::Span<float const> predts, MetaInfo const &info,
             int32_t device, std::shared_ptr<DeviceAUCCache> *p_cache) {
  auto& cache = *p_cache;
  if (!cache) {
    cache.reset(new DeviceAUCCache);
  }
  cache->Init(predts, false, device);

  auto labels = info.labels_.ConstDeviceSpan();
  auto weights = info.weights_.ConstDeviceSpan();
  dh::safe_cuda(hipSetDevice(device));

  CHECK(!labels.empty());
  CHECK_EQ(labels.size(), predts.size());

  /**
   * Create sorted index for each class
   */
  auto d_sorted_idx = dh::ToSpan(cache->sorted_idx);
  dh::ArgSort<false>(predts, d_sorted_idx);

  /**
   * Linear scan
   */
  auto get_weight = GetWeightOp{weights, d_sorted_idx};
  using Pair = thrust::pair<float, float>;
  auto get_fp_tp = [=]__device__(size_t i) {
    size_t idx = d_sorted_idx[i];

    float label = labels[idx];
    float w = get_weight(i);

    float fp = (1.0 - label) * w;
    float tp = label * w;

    return thrust::make_pair(fp, tp);
  };  // NOLINT
  auto d_fptp = dh::ToSpan(cache->fptp);
  dh::LaunchN(d_sorted_idx.size(),
              [=] __device__(size_t i) { d_fptp[i] = get_fp_tp(i); });

  dh::XGBDeviceAllocator<char> alloc;
  auto d_unique_idx = dh::ToSpan(cache->unique_idx);
  dh::Iota(d_unique_idx);

  auto uni_key = dh::MakeTransformIterator<float>(
      thrust::make_counting_iterator(0),
      [=] __device__(size_t i) { return predts[d_sorted_idx[i]]; });
  auto end_unique = thrust::unique_by_key_copy(
      thrust::cuda::par(alloc), uni_key, uni_key + d_sorted_idx.size(),
      dh::tbegin(d_unique_idx), thrust::make_discard_iterator(),
      dh::tbegin(d_unique_idx));
  d_unique_idx = d_unique_idx.subspan(0, end_unique.second - dh::tbegin(d_unique_idx));

  dh::InclusiveScan(
      dh::tbegin(d_fptp), dh::tbegin(d_fptp),
      [=] __device__(Pair const &l, Pair const &r) {
        return thrust::make_pair(l.first + r.first, l.second + r.second);
      },
      d_fptp.size());

  auto d_neg_pos = dh::ToSpan(cache->neg_pos);
  // scatter unique negaive/positive values
  // shift to right by 1 with initial value being 0
  dh::LaunchN(d_unique_idx.size(), [=] __device__(size_t i) {
    if (d_unique_idx[i] == 0) {  // first unique index is 0
      assert(i == 0);
      d_neg_pos[0] = {0, 0};
      return;
    }
    d_neg_pos[d_unique_idx[i]] = d_fptp[d_unique_idx[i] - 1];
    if (i == d_unique_idx.size() - 1) {
      // last one needs to be included, may override above assignment if the last
      // prediction value is distinct from previous one.
      d_neg_pos.back() = d_fptp[d_unique_idx[i] - 1];
      return;
    }
  });

  auto in = dh::MakeTransformIterator<float>(
      thrust::make_counting_iterator(0), [=] __device__(size_t i) {
        float fp, tp;
        float fp_prev, tp_prev;
        if (i == 0) {
          // handle the last element
          thrust::tie(fp, tp) = d_fptp.back();
          thrust::tie(fp_prev, tp_prev) = d_neg_pos[d_unique_idx.back()];
        } else {
          thrust::tie(fp, tp) = d_fptp[d_unique_idx[i] - 1];
          thrust::tie(fp_prev, tp_prev) = d_neg_pos[d_unique_idx[i - 1]];
        }
        return TrapesoidArea(fp_prev, fp, tp_prev, tp);
      });

  Pair last = cache->fptp.back();
  float auc = thrust::reduce(thrust::cuda::par(alloc), in, in + d_unique_idx.size());
  return std::make_tuple(last.first, last.second, auc);
}

void Transpose(common::Span<float const> in, common::Span<float> out, size_t m,
               size_t n, int32_t device) {
  CHECK_EQ(in.size(), out.size());
  CHECK_EQ(in.size(), m * n);
  dh::LaunchN(in.size(), [=] __device__(size_t i) {
    size_t col = i / m;
    size_t row = i % m;
    size_t idx = row * n + col;
    out[i] = in[idx];
  });
}

/**
 * Last index of a group in a CSR style of index pointer.
 */
template <typename Idx>
XGBOOST_DEVICE size_t LastOf(size_t group, common::Span<Idx> indptr) {
  return indptr[group + 1] - 1;
}


float ScaleClasses(common::Span<float> results, common::Span<float> local_area,
                   common::Span<float> fp, common::Span<float> tp,
                   common::Span<float> auc, std::shared_ptr<DeviceAUCCache> cache,
                   size_t n_classes) {
  dh::XGBDeviceAllocator<char> alloc;
  if (rabit::IsDistributed()) {
    CHECK_EQ(dh::CudaGetPointerDevice(results.data()), dh::CurrentDevice());
    cache->reducer->AllReduceSum(results.data(), results.data(), results.size());
  }
  auto reduce_in = dh::MakeTransformIterator<thrust::pair<float, float>>(
      thrust::make_counting_iterator(0), [=] __device__(size_t i) {
        if (local_area[i] > 0) {
          return thrust::make_pair(auc[i] / local_area[i] * tp[i], tp[i]);
        }
        return thrust::make_pair(std::numeric_limits<float>::quiet_NaN(), 0.0f);
      });

  float tp_sum;
  float auc_sum;
  thrust::tie(auc_sum, tp_sum) = thrust::reduce(
      thrust::cuda::par(alloc), reduce_in, reduce_in + n_classes,
      thrust::make_pair(0.0f, 0.0f),
      [=] __device__(auto const &l, auto const &r) {
        return thrust::make_pair(l.first + r.first, l.second + r.second);
      });
  if (tp_sum != 0 && !std::isnan(auc_sum)) {
    auc_sum /= tp_sum;
  } else {
    return std::numeric_limits<float>::quiet_NaN();
  }
  return auc_sum;
}

/**
 * MultiClass implementation is similar to binary classification, except we need to split
 * up each class in all kernels.
 */
float GPUMultiClassAUCOVR(common::Span<float const> predts, MetaInfo const &info,
                          int32_t device, std::shared_ptr<DeviceAUCCache>* p_cache,
                          size_t n_classes) {
  dh::safe_cuda(hipSetDevice(device));
  auto& cache = *p_cache;
  if (!cache) {
    cache.reset(new DeviceAUCCache);
  }
  cache->Init(predts, true, device);

  auto labels = info.labels_.ConstDeviceSpan();
  auto weights = info.weights_.ConstDeviceSpan();

  size_t n_samples = labels.size();

  if (n_samples == 0) {
    dh::TemporaryArray<float> resutls(n_classes * 4, 0.0f);
    auto d_results = dh::ToSpan(resutls);
    dh::LaunchN(n_classes * 4,
                [=] __device__(size_t i) { d_results[i] = 0.0f; });
    auto local_area = d_results.subspan(0, n_classes);
    auto fp = d_results.subspan(n_classes, n_classes);
    auto tp = d_results.subspan(2 * n_classes, n_classes);
    auto auc = d_results.subspan(3 * n_classes, n_classes);
    return ScaleClasses(d_results, local_area, fp, tp, auc, cache, n_classes);
  }

  /**
   * Create sorted index for each class
   */
  auto d_predts_t = dh::ToSpan(cache->predts_t);
  Transpose(predts, d_predts_t, n_samples, n_classes, device);

  dh::TemporaryArray<uint32_t> class_ptr(n_classes + 1, 0);
  auto d_class_ptr = dh::ToSpan(class_ptr);
  dh::LaunchN(n_classes + 1,
              [=] __device__(size_t i) { d_class_ptr[i] = i * n_samples; });
  // no out-of-place sort for thrust, cub sort doesn't accept general iterator. So can't
  // use transform iterator in sorting.
  auto d_sorted_idx = dh::ToSpan(cache->sorted_idx);
  dh::SegmentedArgSort<false>(d_predts_t, d_class_ptr, d_sorted_idx);

  /**
   * Linear scan
   */
  dh::caching_device_vector<float> d_auc(n_classes, 0);
  auto s_d_auc = dh::ToSpan(d_auc);
  auto get_weight = GetWeightOp{weights, d_sorted_idx};
  using Pair = thrust::pair<float, float>;
  auto d_fptp = dh::ToSpan(cache->fptp);
  auto get_fp_tp = [=]__device__(size_t i) {
    size_t idx = d_sorted_idx[i];

    size_t class_id = i / n_samples;
    // labels is a vector of size n_samples.
    float label = labels[idx % n_samples] == class_id;

    float w = get_weight(i % n_samples);
    float fp = (1.0 - label) * w;
    float tp = label * w;
    return thrust::make_pair(fp, tp);
  };  // NOLINT
  dh::LaunchN(d_sorted_idx.size(),
              [=] __device__(size_t i) { d_fptp[i] = get_fp_tp(i); });

  /**
   *  Handle duplicated predictions
   */
  dh::XGBDeviceAllocator<char> alloc;
  auto d_unique_idx = dh::ToSpan(cache->unique_idx);
  dh::Iota(d_unique_idx);
  auto uni_key = dh::MakeTransformIterator<thrust::pair<uint32_t, float>>(
      thrust::make_counting_iterator(0), [=] __device__(size_t i) {
        uint32_t class_id = i / n_samples;
        float predt = d_predts_t[d_sorted_idx[i]];
        return thrust::make_pair(class_id, predt);
      });

  // unique values are sparse, so we need a CSR style indptr
  dh::TemporaryArray<uint32_t> unique_class_ptr(class_ptr.size());
  auto d_unique_class_ptr = dh::ToSpan(unique_class_ptr);
  auto n_uniques = dh::SegmentedUniqueByKey(
      thrust::cuda::par(alloc),
      dh::tbegin(d_class_ptr),
      dh::tend(d_class_ptr),
      uni_key,
      uni_key + d_sorted_idx.size(),
      dh::tbegin(d_unique_idx),
      d_unique_class_ptr.data(),
      dh::tbegin(d_unique_idx),
      thrust::equal_to<thrust::pair<uint32_t, float>>{});
  d_unique_idx = d_unique_idx.subspan(0, n_uniques);

  using Triple = thrust::tuple<uint32_t, float, float>;
  // expand to tuple to include class id
  auto fptp_it_in = dh::MakeTransformIterator<Triple>(
      thrust::make_counting_iterator(0), [=] __device__(size_t i) {
        return thrust::make_tuple(i, d_fptp[i].first, d_fptp[i].second);
      });
  // shrink down to pair
  auto fptp_it_out = thrust::make_transform_output_iterator(
      dh::TypedDiscard<Triple>{}, [d_fptp] __device__(Triple const &t) {
        d_fptp[thrust::get<0>(t)] =
            thrust::make_pair(thrust::get<1>(t), thrust::get<2>(t));
        return t;
      });
  dh::InclusiveScan(
      fptp_it_in, fptp_it_out,
      [=] __device__(Triple const &l, Triple const &r) {
        uint32_t l_cid = thrust::get<0>(l) / n_samples;
        uint32_t r_cid = thrust::get<0>(r) / n_samples;
        if (l_cid != r_cid) {
          return r;
        }

        return Triple(thrust::get<0>(r),
                      thrust::get<1>(l) + thrust::get<1>(r),   // fp
                      thrust::get<2>(l) + thrust::get<2>(r));  // tp
      },
      d_fptp.size());

  // scatter unique FP_PREV/TP_PREV values
  auto d_neg_pos = dh::ToSpan(cache->neg_pos);
  // When dataset is not empty, each class must have at least 1 (unique) sample
  // prediction, so no need to handle special case.
  dh::LaunchN(d_unique_idx.size(), [=] __device__(size_t i) {
    if (d_unique_idx[i] % n_samples == 0) {  // first unique index is 0
      assert(d_unique_idx[i] % n_samples == 0);
      d_neg_pos[d_unique_idx[i]] = {0, 0};   // class_id * n_samples = i
      return;
    }
    uint32_t class_id = d_unique_idx[i] / n_samples;
    d_neg_pos[d_unique_idx[i]] = d_fptp[d_unique_idx[i] - 1];
    if (i == LastOf(class_id, d_unique_class_ptr)) {
      // last one needs to be included.
      size_t last = d_unique_idx[LastOf(class_id, d_unique_class_ptr)];
      d_neg_pos[LastOf(class_id, d_class_ptr)] = d_fptp[last - 1];
      return;
    }
  });

  /**
   * Reduce the result for each class
   */
  auto key_in = dh::MakeTransformIterator<uint32_t>(
      thrust::make_counting_iterator(0), [=] __device__(size_t i) {
        size_t class_id = d_unique_idx[i] / n_samples;
        return class_id;
      });
  auto val_in = dh::MakeTransformIterator<float>(
      thrust::make_counting_iterator(0), [=] __device__(size_t i) {
        size_t class_id = d_unique_idx[i] / n_samples;
        float fp, tp;
        float fp_prev, tp_prev;
        if (i == d_unique_class_ptr[class_id]) {
          // first item is ignored, we use this thread to calculate the last item
          thrust::tie(fp, tp) = d_fptp[class_id * n_samples + (n_samples - 1)];
          thrust::tie(fp_prev, tp_prev) =
              d_neg_pos[d_unique_idx[LastOf(class_id, d_unique_class_ptr)]];
        } else {
          thrust::tie(fp, tp) = d_fptp[d_unique_idx[i] - 1];
          thrust::tie(fp_prev, tp_prev) = d_neg_pos[d_unique_idx[i - 1]];
        }
        float auc = TrapesoidArea(fp_prev, fp, tp_prev, tp);
        return auc;
      });

  thrust::reduce_by_key(thrust::cuda::par(alloc), key_in,
                        key_in + d_unique_idx.size(), val_in,
                        thrust::make_discard_iterator(), d_auc.begin());

  /**
   * Scale the classes with number of samples for each class.
   */
  dh::TemporaryArray<float> resutls(n_classes * 4);
  auto d_results = dh::ToSpan(resutls);
  auto local_area = d_results.subspan(0, n_classes);
  auto fp = d_results.subspan(n_classes, n_classes);
  auto tp = d_results.subspan(2 * n_classes, n_classes);
  auto auc = d_results.subspan(3 * n_classes, n_classes);

  dh::LaunchN(n_classes, [=] __device__(size_t c) {
    auc[c] = s_d_auc[c];
    auto last = d_fptp[n_samples * c + (n_samples - 1)];
    fp[c] = last.first;
    tp[c] = last.second;
    local_area[c] = last.first * last.second;
  });
  return ScaleClasses(d_results, local_area, fp, tp, auc, cache, n_classes);
}

namespace {
struct RankScanItem {
  size_t idx;
  float predt;
  float w;
  bst_group_t group_id;
};
}  // anonymous namespace

std::pair<float, uint32_t>
GPURankingAUC(common::Span<float const> predts, MetaInfo const &info,
              int32_t device, std::shared_ptr<DeviceAUCCache> *p_cache) {
  auto& cache = *p_cache;
  if (!cache) {
    cache.reset(new DeviceAUCCache);
  }
  cache->Init(predts, false, device);

  dh::caching_device_vector<bst_group_t> group_ptr(info.group_ptr_);
  dh::XGBCachingDeviceAllocator<char> alloc;

  auto d_group_ptr = dh::ToSpan(group_ptr);
  /**
   * Validate the dataset
   */
  auto check_it = dh::MakeTransformIterator<size_t>(
      thrust::make_counting_iterator(0),
      [=] __device__(size_t i) { return d_group_ptr[i + 1] - d_group_ptr[i]; });
  size_t n_valid = thrust::count_if(
      thrust::cuda::par(alloc), check_it, check_it + group_ptr.size() - 1,
      [=] __device__(size_t len) { return len >= 3; });
  if (n_valid < info.group_ptr_.size() - 1) {
    InvalidGroupAUC();
  }
  if (n_valid == 0) {
    return std::make_pair(0.0f, 0);
  }

  /**
   * Sort the labels
   */
  auto d_labels = info.labels_.ConstDeviceSpan();

  auto d_sorted_idx = dh::ToSpan(cache->sorted_idx);
  dh::SegmentedArgSort<false>(d_labels, d_group_ptr, d_sorted_idx);

  auto d_weights = info.weights_.ConstDeviceSpan();

  dh::caching_device_vector<size_t> threads_group_ptr(group_ptr.size(), 0);
  auto d_threads_group_ptr = dh::ToSpan(threads_group_ptr);
  // Use max to represent triangle
  auto n_threads = common::SegmentedTrapezoidThreads(
      d_group_ptr, d_threads_group_ptr, std::numeric_limits<size_t>::max());
  // get the coordinate in nested summation
  auto get_i_j = [=]__device__(size_t idx, size_t query_group_idx) {
    auto data_group_begin = d_group_ptr[query_group_idx];
    size_t n_samples = d_group_ptr[query_group_idx + 1] - data_group_begin;
    auto thread_group_begin = d_threads_group_ptr[query_group_idx];
    auto idx_in_thread_group = idx - thread_group_begin;

    size_t i, j;
    common::UnravelTrapeziodIdx(idx_in_thread_group, n_samples, &i, &j);
    // we use global index among all groups for sorted idx, so i, j should also be global
    // index.
    i += data_group_begin;
    j += data_group_begin;
    return thrust::make_pair(i, j);
  };  // NOLINT
  auto in = dh::MakeTransformIterator<RankScanItem>(
      thrust::make_counting_iterator(0), [=] __device__(size_t idx) {
        bst_group_t query_group_idx = dh::SegmentId(d_threads_group_ptr, idx);
        auto data_group_begin = d_group_ptr[query_group_idx];
        size_t n_samples = d_group_ptr[query_group_idx + 1] - data_group_begin;
        if (n_samples < 3) {
          // at least 3 documents are required.
          return RankScanItem{idx, 0, 0, query_group_idx};
        }

        size_t i, j;
        thrust::tie(i, j) = get_i_j(idx, query_group_idx);

        float predt = predts[d_sorted_idx[i]] - predts[d_sorted_idx[j]];
        float w = common::Sqr(d_weights.empty() ? 1.0f : d_weights[query_group_idx]);
        if (predt > 0) {
          predt = 1.0;
        } else if (predt == 0) {
          predt = 0.5;
        } else {
          predt = 0;
        }
        predt *= w;
        return RankScanItem{idx, predt, w, query_group_idx};
      });

  dh::TemporaryArray<float> d_auc(group_ptr.size() - 1);
  auto s_d_auc = dh::ToSpan(d_auc);
  auto out = thrust::make_transform_output_iterator(
      dh::TypedDiscard<RankScanItem>{}, [=] __device__(RankScanItem const &item) -> RankScanItem {
        auto group_id = item.group_id;
        assert(group_id < d_group_ptr.size());
        auto data_group_begin = d_group_ptr[group_id];
        size_t n_samples = d_group_ptr[group_id + 1] - data_group_begin;
        // last item of current group
        if (item.idx == LastOf(group_id, d_threads_group_ptr)) {
          if (item.w > 0) {
            s_d_auc[group_id] = item.predt / item.w;
          } else {
            s_d_auc[group_id] = 0;
          }
        }
        return {};  // discard
      });
  dh::InclusiveScan(
      in, out,
      [] __device__(RankScanItem const &l, RankScanItem const &r) {
        if (l.group_id != r.group_id) {
          return r;
        }
        return RankScanItem{r.idx, l.predt + r.predt, l.w + r.w, l.group_id};
      },
      n_threads);

  /**
   * Scale the AUC with number of items in each group.
   */
  float auc = thrust::reduce(thrust::cuda::par(alloc), dh::tbegin(s_d_auc),
                             dh::tend(s_d_auc), 0.0f);
  return std::make_pair(auc, n_valid);
}
}  // namespace metric
}  // namespace xgboost
