#include "hip/hip_runtime.h"
/*!
 * Copyright 2021-2022 by XGBoost Contributors
 */
#include <thrust/scan.h>
#include <hipcub/hipcub.hpp>

#include <algorithm>
#include <cassert>
#include <limits>
#include <memory>
#include <utility>
#include <tuple>

#include "rabit/rabit.h"
#include "xgboost/span.h"
#include "xgboost/data.h"
#include "auc.h"
#include "../common/device_helpers.cuh"
#include "../common/ranking_utils.cuh"

namespace xgboost {
namespace metric {
namespace {
// Pair of FP/TP
using Pair = thrust::pair<double, double>;

template <typename T, typename U, typename P = thrust::pair<T, U>>
struct PairPlus : public thrust::binary_function<P, P, P> {
  XGBOOST_DEVICE P operator()(P const& l, P const& r) const {
    return thrust::make_pair(l.first + r.first, l.second + r.second);
  }
};
}  // namespace

/**
 * A cache to GPU data to avoid reallocating memory.
 */
struct DeviceAUCCache {
  // index sorted by prediction value
  dh::device_vector<size_t> sorted_idx;
  // track FP/TP for computation on trapezoid area
  dh::device_vector<Pair> fptp;
  // track FP_PREV/TP_PREV for computation on trapezoid area
  dh::device_vector<Pair> neg_pos;
  // index of unique prediction values.
  dh::device_vector<size_t> unique_idx;
  // p^T: transposed prediction matrix, used by MultiClassAUC
  dh::device_vector<float> predts_t;
  std::unique_ptr<dh::AllReducer> reducer;

  void Init(common::Span<float const> predts, bool is_multi, int32_t device) {
    if (sorted_idx.size() != predts.size()) {
      sorted_idx.resize(predts.size());
      fptp.resize(sorted_idx.size());
      unique_idx.resize(sorted_idx.size());
      neg_pos.resize(sorted_idx.size());
      if (is_multi) {
        predts_t.resize(sorted_idx.size());
      }
    }
    if (is_multi && !reducer) {
      reducer.reset(new dh::AllReducer);
      reducer->Init(device);
    }
  }
};

template <bool is_multi>
void InitCacheOnce(common::Span<float const> predts, int32_t device,
                   std::shared_ptr<DeviceAUCCache>* p_cache) {
  auto& cache = *p_cache;
  if (!cache) {
    cache.reset(new DeviceAUCCache);
  }
  cache->Init(predts, is_multi, device);
}

/**
 * The GPU implementation uses same calculation as CPU with a few more steps to distribute
 * work across threads:
 *
 * - Run scan to obtain TP/FP values, which are right coordinates of trapezoid.
 * - Find distinct prediction values and get the corresponding FP_PREV/TP_PREV value,
 *   which are left coordinates of trapezoids.
 * - Reduce the scan array into 1 AUC value.
 */
template <typename Fn>
std::tuple<double, double, double>
GPUBinaryAUC(common::Span<float const> predts, MetaInfo const &info,
             int32_t device, common::Span<size_t const> d_sorted_idx,
             Fn area_fn, std::shared_ptr<DeviceAUCCache> cache) {
  auto labels = info.labels.View(device);
  auto weights = info.weights_.ConstDeviceSpan();
  dh::safe_cuda(hipSetDevice(device));

  CHECK_NE(labels.Size(), 0);
  CHECK_EQ(labels.Size(), predts.size());

  /**
   * Linear scan
   */
  auto get_weight = common::OptionalWeights{weights};
  auto get_fp_tp = [=]XGBOOST_DEVICE(size_t i) {
    size_t idx = d_sorted_idx[i];

    float label = labels(idx);
    float w = get_weight[d_sorted_idx[i]];

    float fp = (1.0 - label) * w;
    float tp = label * w;

    return thrust::make_pair(fp, tp);
  };  // NOLINT
  auto d_fptp = dh::ToSpan(cache->fptp);
  dh::LaunchN(d_sorted_idx.size(),
              [=] XGBOOST_DEVICE(size_t i) { d_fptp[i] = get_fp_tp(i); });

  dh::XGBDeviceAllocator<char> alloc;
  auto d_unique_idx = dh::ToSpan(cache->unique_idx);
  dh::Iota(d_unique_idx);

  auto uni_key = dh::MakeTransformIterator<float>(
      thrust::make_counting_iterator(0),
      [=] XGBOOST_DEVICE(size_t i) { return predts[d_sorted_idx[i]]; });
  auto end_unique = thrust::unique_by_key_copy(
      thrust::cuda::par(alloc), uni_key, uni_key + d_sorted_idx.size(),
      dh::tbegin(d_unique_idx), thrust::make_discard_iterator(),
      dh::tbegin(d_unique_idx));
  d_unique_idx = d_unique_idx.subspan(0, end_unique.second - dh::tbegin(d_unique_idx));

  dh::InclusiveScan(dh::tbegin(d_fptp), dh::tbegin(d_fptp),
                    PairPlus<double, double>{}, d_fptp.size());

  auto d_neg_pos = dh::ToSpan(cache->neg_pos);
  // scatter unique negaive/positive values
  // shift to right by 1 with initial value being 0
  dh::LaunchN(d_unique_idx.size(), [=] XGBOOST_DEVICE(size_t i) {
    if (d_unique_idx[i] == 0) {  // first unique index is 0
      assert(i == 0);
      d_neg_pos[0] = {0, 0};
      return;
    }
    d_neg_pos[d_unique_idx[i]] = d_fptp[d_unique_idx[i] - 1];
    if (i == d_unique_idx.size() - 1) {
      // last one needs to be included, may override above assignment if the last
      // prediction value is distinct from previous one.
      d_neg_pos.back() = d_fptp[d_unique_idx[i] - 1];
      return;
    }
  });

  auto in = dh::MakeTransformIterator<double>(
      thrust::make_counting_iterator(0), [=] XGBOOST_DEVICE(size_t i) {
        double fp, tp;
        double fp_prev, tp_prev;
        if (i == 0) {
          // handle the last element
          thrust::tie(fp, tp) = d_fptp.back();
          thrust::tie(fp_prev, tp_prev) = d_neg_pos[d_unique_idx.back()];
        } else {
          thrust::tie(fp, tp) = d_fptp[d_unique_idx[i] - 1];
          thrust::tie(fp_prev, tp_prev) = d_neg_pos[d_unique_idx[i - 1]];
        }
        return area_fn(fp_prev, fp, tp_prev, tp);
      });

  Pair last = cache->fptp.back();
  double auc = thrust::reduce(thrust::cuda::par(alloc), in, in + d_unique_idx.size());
  return std::make_tuple(last.first, last.second, auc);
}

std::tuple<double, double, double>
GPUBinaryROCAUC(common::Span<float const> predts, MetaInfo const &info,
                int32_t device, std::shared_ptr<DeviceAUCCache> *p_cache) {
  auto &cache = *p_cache;
  InitCacheOnce<false>(predts, device, p_cache);

  /**
   * Create sorted index for each class
   */
  auto d_sorted_idx = dh::ToSpan(cache->sorted_idx);
  dh::ArgSort<false>(predts, d_sorted_idx);
  // Create lambda to avoid pass function pointer.
  return GPUBinaryAUC(
      predts, info, device, d_sorted_idx,
      [] XGBOOST_DEVICE(double x0, double x1, double y0, double y1) -> double {
        return TrapezoidArea(x0, x1, y0, y1);
      },
      cache);
}

void Transpose(common::Span<float const> in, common::Span<float> out, size_t m,
               size_t n) {
  CHECK_EQ(in.size(), out.size());
  CHECK_EQ(in.size(), m * n);
  dh::LaunchN(in.size(), [=] XGBOOST_DEVICE(size_t i) {
    size_t col = i / m;
    size_t row = i % m;
    size_t idx = row * n + col;
    out[i] = in[idx];
  });
}

double ScaleClasses(common::Span<double> results, common::Span<double> local_area,
                    common::Span<double> tp, common::Span<double> auc,
                    std::shared_ptr<DeviceAUCCache> cache, size_t n_classes) {
  dh::XGBDeviceAllocator<char> alloc;
  if (rabit::IsDistributed()) {
    CHECK_EQ(dh::CudaGetPointerDevice(results.data()), dh::CurrentDevice());
    cache->reducer->AllReduceSum(results.data(), results.data(), results.size());
  }
  auto reduce_in = dh::MakeTransformIterator<Pair>(
      thrust::make_counting_iterator(0), [=] XGBOOST_DEVICE(size_t i) {
        if (local_area[i] > 0) {
          return thrust::make_pair(auc[i] / local_area[i] * tp[i], tp[i]);
        }
        return thrust::make_pair(std::numeric_limits<double>::quiet_NaN(), 0.0);
      });

  double tp_sum;
  double auc_sum;
  thrust::tie(auc_sum, tp_sum) =
      thrust::reduce(thrust::cuda::par(alloc), reduce_in, reduce_in + n_classes,
                     Pair{0.0, 0.0}, PairPlus<double, double>{});
  if (tp_sum != 0 && !std::isnan(auc_sum)) {
    auc_sum /= tp_sum;
  } else {
    return std::numeric_limits<double>::quiet_NaN();
  }
  return auc_sum;
}

/**
 * Calculate FP/TP for multi-class and PR-AUC ranking. `segment_id` is a function for
 * getting class id or group id given scan index.
 */
template <typename Fn>
void SegmentedFPTP(common::Span<Pair> d_fptp, Fn segment_id) {
  using Triple = thrust::tuple<uint32_t, double, double>;
  // expand to tuple to include idx
  auto fptp_it_in = dh::MakeTransformIterator<Triple>(
      thrust::make_counting_iterator(0), [=] XGBOOST_DEVICE(size_t i) {
        return thrust::make_tuple(i, d_fptp[i].first, d_fptp[i].second);
      });
  // shrink down to pair
  auto fptp_it_out = thrust::make_transform_output_iterator(
      dh::TypedDiscard<Triple>{}, [d_fptp] XGBOOST_DEVICE(Triple const &t) {
        d_fptp[thrust::get<0>(t)] =
            thrust::make_pair(thrust::get<1>(t), thrust::get<2>(t));
        return t;
      });
  dh::InclusiveScan(
      fptp_it_in, fptp_it_out,
      [=] XGBOOST_DEVICE(Triple const &l, Triple const &r) {
        uint32_t l_gid = segment_id(thrust::get<0>(l));
        uint32_t r_gid = segment_id(thrust::get<0>(r));
        if (l_gid != r_gid) {
          return r;
        }

        return Triple(thrust::get<0>(r),
                      thrust::get<1>(l) + thrust::get<1>(r),   // fp
                      thrust::get<2>(l) + thrust::get<2>(r));  // tp
      },
      d_fptp.size());
}

/**
 * Reduce the values of AUC for each group/class.
 */
template <typename Area, typename Seg>
void SegmentedReduceAUC(common::Span<size_t const> d_unique_idx,
                        common::Span<uint32_t const> d_class_ptr,
                        common::Span<uint32_t const> d_unique_class_ptr,
                        std::shared_ptr<DeviceAUCCache> cache,
                        Area area_fn,
                        Seg segment_id,
                        common::Span<double> d_auc) {
  auto d_fptp = dh::ToSpan(cache->fptp);
  auto d_neg_pos = dh::ToSpan(cache->neg_pos);
  dh::XGBDeviceAllocator<char> alloc;
  auto key_in = dh::MakeTransformIterator<uint32_t>(
      thrust::make_counting_iterator(0), [=] XGBOOST_DEVICE(size_t i) {
        size_t class_id = segment_id(d_unique_idx[i]);
        return class_id;
      });
  auto val_in = dh::MakeTransformIterator<double>(
      thrust::make_counting_iterator(0), [=] XGBOOST_DEVICE(size_t i) {
        size_t class_id = segment_id(d_unique_idx[i]);

        double fp, tp, fp_prev, tp_prev;
        if (i == d_unique_class_ptr[class_id]) {
          // first item is ignored, we use this thread to calculate the last item
          thrust::tie(fp, tp) = d_fptp[common::LastOf(class_id, d_class_ptr)];
          thrust::tie(fp_prev, tp_prev) =
              d_neg_pos[d_unique_idx[common::LastOf(class_id, d_unique_class_ptr)]];
        } else {
          thrust::tie(fp, tp) = d_fptp[d_unique_idx[i] - 1];
          thrust::tie(fp_prev, tp_prev) = d_neg_pos[d_unique_idx[i - 1]];
        }
        double auc = area_fn(fp_prev, fp, tp_prev, tp, class_id);
        return auc;
      });
  thrust::reduce_by_key(thrust::cuda::par(alloc), key_in,
                        key_in + d_unique_idx.size(), val_in,
                        thrust::make_discard_iterator(), dh::tbegin(d_auc));
}

/**
 * MultiClass implementation is similar to binary classification, except we need to split
 * up each class in all kernels.
 */
template <bool scale, typename Fn>
double GPUMultiClassAUCOVR(MetaInfo const &info, int32_t device, common::Span<uint32_t> d_class_ptr,
                           size_t n_classes, std::shared_ptr<DeviceAUCCache> cache, Fn area_fn) {
  dh::safe_cuda(hipSetDevice(device));
  /**
   * Sorted idx
   */
  auto d_predts_t = dh::ToSpan(cache->predts_t);
  // Index is sorted within class.
  auto d_sorted_idx = dh::ToSpan(cache->sorted_idx);

  auto labels = info.labels.View(device);
  auto weights = info.weights_.ConstDeviceSpan();

  size_t n_samples = labels.Shape(0);

  if (n_samples == 0) {
    dh::TemporaryArray<double> resutls(n_classes * 4, 0.0f);
    auto d_results = dh::ToSpan(resutls);
    dh::LaunchN(n_classes * 4,
                [=] XGBOOST_DEVICE(size_t i) { d_results[i] = 0.0f; });
    auto local_area = d_results.subspan(0, n_classes);
    auto tp = d_results.subspan(2 * n_classes, n_classes);
    auto auc = d_results.subspan(3 * n_classes, n_classes);
    return ScaleClasses(d_results, local_area, tp, auc, cache, n_classes);
  }

  /**
   * Linear scan
   */
  dh::caching_device_vector<double> d_auc(n_classes, 0);
  auto get_weight = common::OptionalWeights{weights};
  auto d_fptp = dh::ToSpan(cache->fptp);
  auto get_fp_tp = [=]XGBOOST_DEVICE(size_t i) {
    size_t idx = d_sorted_idx[i];

    size_t class_id = i / n_samples;
    // labels is a vector of size n_samples.
    float label = labels(idx % n_samples) == class_id;

    float w = get_weight[d_sorted_idx[i] % n_samples];
    float fp = (1.0 - label) * w;
    float tp = label * w;
    return thrust::make_pair(fp, tp);
  };  // NOLINT
  dh::LaunchN(d_sorted_idx.size(),
              [=] XGBOOST_DEVICE(size_t i) { d_fptp[i] = get_fp_tp(i); });

  /**
   *  Handle duplicated predictions
   */
  dh::XGBDeviceAllocator<char> alloc;
  auto d_unique_idx = dh::ToSpan(cache->unique_idx);
  dh::Iota(d_unique_idx);
  auto uni_key = dh::MakeTransformIterator<thrust::pair<uint32_t, float>>(
      thrust::make_counting_iterator(0), [=] XGBOOST_DEVICE(size_t i) {
        uint32_t class_id = i / n_samples;
        float predt = d_predts_t[d_sorted_idx[i]];
        return thrust::make_pair(class_id, predt);
      });

  // unique values are sparse, so we need a CSR style indptr
  dh::TemporaryArray<uint32_t> unique_class_ptr(d_class_ptr.size());
  auto d_unique_class_ptr = dh::ToSpan(unique_class_ptr);
  auto n_uniques = dh::SegmentedUniqueByKey(
      thrust::cuda::par(alloc),
      dh::tbegin(d_class_ptr),
      dh::tend(d_class_ptr),
      uni_key,
      uni_key + d_sorted_idx.size(),
      dh::tbegin(d_unique_idx),
      d_unique_class_ptr.data(),
      dh::tbegin(d_unique_idx),
      thrust::equal_to<thrust::pair<uint32_t, float>>{});
  d_unique_idx = d_unique_idx.subspan(0, n_uniques);

  auto get_class_id = [=] XGBOOST_DEVICE(size_t idx) { return idx / n_samples; };
  SegmentedFPTP(d_fptp, get_class_id);

  // scatter unique FP_PREV/TP_PREV values
  auto d_neg_pos = dh::ToSpan(cache->neg_pos);
  // When dataset is not empty, each class must have at least 1 (unique) sample
  // prediction, so no need to handle special case.
  dh::LaunchN(d_unique_idx.size(), [=] XGBOOST_DEVICE(size_t i) {
    if (d_unique_idx[i] % n_samples == 0) {  // first unique index is 0
      assert(d_unique_idx[i] % n_samples == 0);
      d_neg_pos[d_unique_idx[i]] = {0, 0};   // class_id * n_samples = i
      return;
    }
    uint32_t class_id = d_unique_idx[i] / n_samples;
    d_neg_pos[d_unique_idx[i]] = d_fptp[d_unique_idx[i] - 1];
    if (i == common::LastOf(class_id, d_unique_class_ptr)) {
      // last one needs to be included.
      size_t last = d_unique_idx[common::LastOf(class_id, d_unique_class_ptr)];
      d_neg_pos[common::LastOf(class_id, d_class_ptr)] = d_fptp[last - 1];
      return;
    }
  });

  /**
   * Reduce the result for each class
   */
  auto s_d_auc = dh::ToSpan(d_auc);
  SegmentedReduceAUC(d_unique_idx, d_class_ptr, d_unique_class_ptr, cache,
                     area_fn, get_class_id, s_d_auc);

  /**
   * Scale the classes with number of samples for each class.
   */
  dh::TemporaryArray<double> resutls(n_classes * 4);
  auto d_results = dh::ToSpan(resutls);
  auto local_area = d_results.subspan(0, n_classes);
  auto fp = d_results.subspan(n_classes, n_classes);
  auto tp = d_results.subspan(2 * n_classes, n_classes);
  auto auc = d_results.subspan(3 * n_classes, n_classes);

  dh::LaunchN(n_classes, [=] XGBOOST_DEVICE(size_t c) {
    auc[c] = s_d_auc[c];
    auto last = d_fptp[n_samples * c + (n_samples - 1)];
    fp[c] = last.first;
    if (scale) {
      local_area[c] = last.first * last.second;
      tp[c] = last.second;
    } else {
      local_area[c] = 1.0f;
      tp[c] = 1.0f;
    }
  });
  return ScaleClasses(d_results, local_area, tp, auc, cache, n_classes);
}

void MultiClassSortedIdx(common::Span<float const> predts,
                         common::Span<uint32_t> d_class_ptr,
                         std::shared_ptr<DeviceAUCCache> cache) {
  size_t n_classes = d_class_ptr.size() - 1;
  auto d_predts_t = dh::ToSpan(cache->predts_t);
  auto n_samples = d_predts_t.size() / n_classes;
  if (n_samples == 0) {
    return;
  }
  Transpose(predts, d_predts_t, n_samples, n_classes);
  dh::LaunchN(n_classes + 1,
              [=] XGBOOST_DEVICE(size_t i) { d_class_ptr[i] = i * n_samples; });
  auto d_sorted_idx = dh::ToSpan(cache->sorted_idx);
  dh::SegmentedArgSort<false>(d_predts_t, d_class_ptr, d_sorted_idx);
}

double GPUMultiClassROCAUC(common::Span<float const> predts,
                           MetaInfo const &info, int32_t device,
                           std::shared_ptr<DeviceAUCCache> *p_cache,
                           size_t n_classes) {
  auto& cache = *p_cache;
  InitCacheOnce<true>(predts, device, p_cache);

  /**
   * Create sorted index for each class
   */
  dh::TemporaryArray<uint32_t> class_ptr(n_classes + 1, 0);
  MultiClassSortedIdx(predts, dh::ToSpan(class_ptr), cache);

  auto fn = [] XGBOOST_DEVICE(double fp_prev, double fp, double tp_prev,
                              double tp, size_t /*class_id*/) {
    return TrapezoidArea(fp_prev, fp, tp_prev, tp);
  };
  return GPUMultiClassAUCOVR<true>(info, device, dh::ToSpan(class_ptr), n_classes, cache, fn);
}

namespace {
struct RankScanItem {
  size_t idx;
  double predt;
  double w;
  bst_group_t group_id;
};
}  // anonymous namespace

std::pair<double, uint32_t>
GPURankingAUC(common::Span<float const> predts, MetaInfo const &info,
              int32_t device, std::shared_ptr<DeviceAUCCache> *p_cache) {
  auto& cache = *p_cache;
  InitCacheOnce<false>(predts, device, p_cache);

  dh::caching_device_vector<bst_group_t> group_ptr(info.group_ptr_);
  dh::XGBCachingDeviceAllocator<char> alloc;

  auto d_group_ptr = dh::ToSpan(group_ptr);
  /**
   * Validate the dataset
   */
  auto check_it = dh::MakeTransformIterator<size_t>(
      thrust::make_counting_iterator(0),
      [=] XGBOOST_DEVICE(size_t i) { return d_group_ptr[i + 1] - d_group_ptr[i]; });
  size_t n_valid = thrust::count_if(
      thrust::cuda::par(alloc), check_it, check_it + group_ptr.size() - 1,
      [=] XGBOOST_DEVICE(size_t len) { return len >= 3; });
  if (n_valid < info.group_ptr_.size() - 1) {
    InvalidGroupAUC();
  }
  if (n_valid == 0) {
    return std::make_pair(0.0, 0);
  }

  /**
   * Sort the labels
   */
  auto d_labels = info.labels.View(device);

  auto d_sorted_idx = dh::ToSpan(cache->sorted_idx);
  dh::SegmentedArgSort<false>(d_labels.Values(), d_group_ptr, d_sorted_idx);

  auto d_weights = info.weights_.ConstDeviceSpan();

  dh::caching_device_vector<size_t> threads_group_ptr(group_ptr.size(), 0);
  auto d_threads_group_ptr = dh::ToSpan(threads_group_ptr);
  // Use max to represent triangle
  auto n_threads = common::SegmentedTrapezoidThreads(
      d_group_ptr, d_threads_group_ptr, std::numeric_limits<size_t>::max());
  CHECK_LT(n_threads, std::numeric_limits<int32_t>::max());
  // get the coordinate in nested summation
  auto get_i_j = [=]XGBOOST_DEVICE(size_t idx, size_t query_group_idx) {
    auto data_group_begin = d_group_ptr[query_group_idx];
    size_t n_samples = d_group_ptr[query_group_idx + 1] - data_group_begin;
    auto thread_group_begin = d_threads_group_ptr[query_group_idx];
    auto idx_in_thread_group = idx - thread_group_begin;

    size_t i, j;
    common::UnravelTrapeziodIdx(idx_in_thread_group, n_samples, &i, &j);
    // we use global index among all groups for sorted idx, so i, j should also be global
    // index.
    i += data_group_begin;
    j += data_group_begin;
    return thrust::make_pair(i, j);
  };  // NOLINT
  auto in = dh::MakeTransformIterator<RankScanItem>(
      thrust::make_counting_iterator(0), [=] XGBOOST_DEVICE(size_t idx) {
        bst_group_t query_group_idx = dh::SegmentId(d_threads_group_ptr, idx);
        auto data_group_begin = d_group_ptr[query_group_idx];
        size_t n_samples = d_group_ptr[query_group_idx + 1] - data_group_begin;
        if (n_samples < 3) {
          // at least 3 documents are required.
          return RankScanItem{idx, 0, 0, query_group_idx};
        }

        size_t i, j;
        thrust::tie(i, j) = get_i_j(idx, query_group_idx);

        float predt = predts[d_sorted_idx[i]] - predts[d_sorted_idx[j]];
        float w = common::Sqr(d_weights.empty() ? 1.0f : d_weights[query_group_idx]);
        if (predt > 0) {
          predt = 1.0;
        } else if (predt == 0) {
          predt = 0.5;
        } else {
          predt = 0;
        }
        predt *= w;
        return RankScanItem{idx, predt, w, query_group_idx};
      });

  dh::TemporaryArray<double> d_auc(group_ptr.size() - 1);
  auto s_d_auc = dh::ToSpan(d_auc);
  auto out = thrust::make_transform_output_iterator(
      dh::TypedDiscard<RankScanItem>{},
      [=] XGBOOST_DEVICE(RankScanItem const &item) -> RankScanItem {
        auto group_id = item.group_id;
        assert(group_id < d_group_ptr.size());
        auto data_group_begin = d_group_ptr[group_id];
        size_t n_samples = d_group_ptr[group_id + 1] - data_group_begin;
        // last item of current group
        if (item.idx == common::LastOf(group_id, d_threads_group_ptr)) {
          if (item.w > 0) {
            s_d_auc[group_id] = item.predt / item.w;
          } else {
            s_d_auc[group_id] = 0;
          }
        }
        return {};  // discard
      });
  dh::InclusiveScan(
      in, out,
      [] XGBOOST_DEVICE(RankScanItem const &l, RankScanItem const &r) {
        if (l.group_id != r.group_id) {
          return r;
        }
        return RankScanItem{r.idx, l.predt + r.predt, l.w + r.w, l.group_id};
      },
      n_threads);

  /**
   * Scale the AUC with number of items in each group.
   */
  double auc = thrust::reduce(thrust::cuda::par(alloc), dh::tbegin(s_d_auc),
                              dh::tend(s_d_auc), 0.0);
  return std::make_pair(auc, n_valid);
}

std::tuple<double, double, double>
GPUBinaryPRAUC(common::Span<float const> predts, MetaInfo const &info,
               int32_t device, std::shared_ptr<DeviceAUCCache> *p_cache) {
  auto& cache = *p_cache;
  InitCacheOnce<false>(predts, device, p_cache);

  /**
   * Create sorted index for each class
   */
  auto d_sorted_idx = dh::ToSpan(cache->sorted_idx);
  dh::ArgSort<false>(predts, d_sorted_idx);

  auto labels = info.labels.View(device);
  auto d_weights = info.weights_.ConstDeviceSpan();
  auto get_weight = common::OptionalWeights{d_weights};
  auto it = dh::MakeTransformIterator<Pair>(
      thrust::make_counting_iterator(0ul), [=] XGBOOST_DEVICE(size_t i) {
        auto w = get_weight[d_sorted_idx[i]];
        return thrust::make_pair(labels(d_sorted_idx[i]) * w,
                                 (1.0f - labels(d_sorted_idx[i])) * w);
      });
  dh::XGBCachingDeviceAllocator<char> alloc;
  double total_pos, total_neg;
  thrust::tie(total_pos, total_neg) =
      thrust::reduce(thrust::cuda::par(alloc), it, it + labels.Size(),
                     Pair{0.0, 0.0}, PairPlus<double, double>{});

  if (total_pos <= 0.0 || total_neg <= 0.0) {
    return {0.0f, 0.0f, 0.0f};
  }

  auto fn = [total_pos] XGBOOST_DEVICE(double fp_prev, double fp, double tp_prev,
                                       double tp) {
    return detail::CalcDeltaPRAUC(fp_prev, fp, tp_prev, tp, total_pos);
  };
  double fp, tp, auc;
  std::tie(fp, tp, auc) = GPUBinaryAUC(predts, info, device, d_sorted_idx, fn, cache);
  return std::make_tuple(1.0, 1.0, auc);
}

double GPUMultiClassPRAUC(common::Span<float const> predts,
                          MetaInfo const &info, int32_t device,
                          std::shared_ptr<DeviceAUCCache> *p_cache,
                          size_t n_classes) {
  auto& cache = *p_cache;
  InitCacheOnce<true>(predts, device, p_cache);

  /**
   * Create sorted index for each class
   */
  dh::TemporaryArray<uint32_t> class_ptr(n_classes + 1, 0);
  auto d_class_ptr = dh::ToSpan(class_ptr);
  MultiClassSortedIdx(predts, d_class_ptr, cache);
  auto d_sorted_idx = dh::ToSpan(cache->sorted_idx);

  auto d_weights = info.weights_.ConstDeviceSpan();

  /**
   * Get total positive/negative
   */
  auto labels = info.labels.View(device);
  auto n_samples = info.num_row_;
  dh::caching_device_vector<Pair> totals(n_classes);
  auto key_it =
      dh::MakeTransformIterator<size_t>(thrust::make_counting_iterator(0ul),
                                        [n_samples] XGBOOST_DEVICE(size_t i) {
                                          return i / n_samples;  // class id
                                        });
  auto get_weight = common::OptionalWeights{d_weights};
  auto val_it = dh::MakeTransformIterator<thrust::pair<double, double>>(
      thrust::make_counting_iterator(0ul), [=] XGBOOST_DEVICE(size_t i) {
        auto idx = d_sorted_idx[i] % n_samples;
        auto w = get_weight[idx];
        auto class_id = i / n_samples;
        auto y = labels(idx) == class_id;
        return thrust::make_pair(y * w, (1.0f - y) * w);
      });
  dh::XGBCachingDeviceAllocator<char> alloc;
  thrust::reduce_by_key(thrust::cuda::par(alloc), key_it,
                        key_it + predts.size(), val_it,
                        thrust::make_discard_iterator(), totals.begin(),
                        thrust::equal_to<size_t>{}, PairPlus<double, double>{});

  /**
   * Calculate AUC
   */
  auto d_totals = dh::ToSpan(totals);
  auto fn = [d_totals] XGBOOST_DEVICE(double fp_prev, double fp, double tp_prev,
                                      double tp, size_t class_id) {
    auto total_pos = d_totals[class_id].first;
    return detail::CalcDeltaPRAUC(fp_prev, fp, tp_prev, tp,
                                  d_totals[class_id].first);
  };
  return GPUMultiClassAUCOVR<false>(info, device, d_class_ptr, n_classes, cache, fn);
}

template <typename Fn>
std::pair<double, uint32_t>
GPURankingPRAUCImpl(common::Span<float const> predts, MetaInfo const &info,
                    common::Span<uint32_t> d_group_ptr, int32_t device,
                    std::shared_ptr<DeviceAUCCache> cache, Fn area_fn) {
  /**
   * Sorted idx
   */
  auto d_sorted_idx = dh::ToSpan(cache->sorted_idx);

  auto labels = info.labels.View(device);
  auto weights = info.weights_.ConstDeviceSpan();

  uint32_t n_groups = static_cast<uint32_t>(info.group_ptr_.size() - 1);

  /**
   * Linear scan
   */
  size_t n_samples = labels.Shape(0);
  dh::caching_device_vector<double> d_auc(n_groups, 0);
  auto get_weight = common::OptionalWeights{weights};
  auto d_fptp = dh::ToSpan(cache->fptp);
  auto get_fp_tp = [=] XGBOOST_DEVICE(size_t i) {
    size_t idx = d_sorted_idx[i];

    size_t group_id = dh::SegmentId(d_group_ptr, idx);
    float label = labels(idx);

    float w = get_weight[group_id];
    float fp = (1.0 - label) * w;
    float tp = label * w;
    return thrust::make_pair(fp, tp);
  };  // NOLINT
  dh::LaunchN(d_sorted_idx.size(),
              [=] XGBOOST_DEVICE(size_t i) { d_fptp[i] = get_fp_tp(i); });

  /**
   *  Handle duplicated predictions
   */
  dh::XGBDeviceAllocator<char> alloc;
  auto d_unique_idx = dh::ToSpan(cache->unique_idx);
  dh::Iota(d_unique_idx);
  auto uni_key = dh::MakeTransformIterator<thrust::pair<uint32_t, float>>(
      thrust::make_counting_iterator(0), [=] XGBOOST_DEVICE(size_t i) {
        auto idx = d_sorted_idx[i];
        bst_group_t group_id = dh::SegmentId(d_group_ptr, idx);
        float predt = predts[idx];
        return thrust::make_pair(group_id, predt);
      });

  // unique values are sparse, so we need a CSR style indptr
  dh::TemporaryArray<uint32_t> unique_class_ptr(d_group_ptr.size());
  auto d_unique_class_ptr = dh::ToSpan(unique_class_ptr);
  auto n_uniques = dh::SegmentedUniqueByKey(
      thrust::cuda::par(alloc),
      dh::tbegin(d_group_ptr),
      dh::tend(d_group_ptr),
      uni_key,
      uni_key + d_sorted_idx.size(),
      dh::tbegin(d_unique_idx),
      d_unique_class_ptr.data(),
      dh::tbegin(d_unique_idx),
      thrust::equal_to<thrust::pair<uint32_t, float>>{});
  d_unique_idx = d_unique_idx.subspan(0, n_uniques);

  auto get_group_id = [=] XGBOOST_DEVICE(size_t idx) {
    return dh::SegmentId(d_group_ptr, idx);
  };
  SegmentedFPTP(d_fptp, get_group_id);

  // scatter unique FP_PREV/TP_PREV values
  auto d_neg_pos = dh::ToSpan(cache->neg_pos);
  dh::LaunchN(d_unique_idx.size(), [=] XGBOOST_DEVICE(size_t i) {
    if (thrust::binary_search(thrust::seq, d_unique_class_ptr.cbegin(),
                              d_unique_class_ptr.cend(),
                              i)) {  // first unique index is 0
      d_neg_pos[d_unique_idx[i]] = {0, 0};
      return;
    }
    auto group_idx = dh::SegmentId(d_group_ptr, d_unique_idx[i]);
    d_neg_pos[d_unique_idx[i]] = d_fptp[d_unique_idx[i] - 1];
    if (i == common::LastOf(group_idx, d_unique_class_ptr)) {
      // last one needs to be included.
      size_t last = d_unique_idx[common::LastOf(group_idx, d_unique_class_ptr)];
      d_neg_pos[common::LastOf(group_idx, d_group_ptr)] = d_fptp[last - 1];
      return;
    }
  });

  /**
   * Reduce the result for each group
   */
  auto s_d_auc = dh::ToSpan(d_auc);
  SegmentedReduceAUC(d_unique_idx, d_group_ptr, d_unique_class_ptr, cache,
                     area_fn, get_group_id, s_d_auc);

  /**
   * Scale the groups with number of samples for each group.
   */
  double auc;
  uint32_t invalid_groups;
  {
    auto it = dh::MakeTransformIterator<thrust::pair<double, uint32_t>>(
        thrust::make_counting_iterator(0ul), [=] XGBOOST_DEVICE(size_t g) {
          double fp, tp;
          thrust::tie(fp, tp) = d_fptp[common::LastOf(g, d_group_ptr)];
          double area = fp * tp;
          auto n_documents = d_group_ptr[g + 1] - d_group_ptr[g];
          if (area > 0 && n_documents >= 2) {
            return thrust::make_pair(s_d_auc[g], static_cast<uint32_t>(0));
          }
          return thrust::make_pair(0.0, static_cast<uint32_t>(1));
        });
    thrust::tie(auc, invalid_groups) = thrust::reduce(
        thrust::cuda::par(alloc), it, it + n_groups,
        thrust::pair<double, uint32_t>(0.0, 0), PairPlus<double, uint32_t>{});
  }
  return std::make_pair(auc, n_groups - invalid_groups);
}

std::pair<double, uint32_t>
GPURankingPRAUC(common::Span<float const> predts, MetaInfo const &info,
                int32_t device, std::shared_ptr<DeviceAUCCache> *p_cache) {
  dh::safe_cuda(hipSetDevice(device));
  if (predts.empty()) {
    return std::make_pair(0.0, static_cast<uint32_t>(0));
  }

  auto &cache = *p_cache;
  InitCacheOnce<false>(predts, device, p_cache);

  dh::device_vector<bst_group_t> group_ptr(info.group_ptr_.size());
  thrust::copy(info.group_ptr_.begin(), info.group_ptr_.end(), group_ptr.begin());
  auto d_group_ptr = dh::ToSpan(group_ptr);
  CHECK_GE(info.group_ptr_.size(), 1) << "Must have at least 1 query group for LTR.";
  size_t n_groups = info.group_ptr_.size() - 1;

  /**
   * Create sorted index for each group
   */
  auto d_sorted_idx = dh::ToSpan(cache->sorted_idx);
  dh::SegmentedArgSort<false>(predts, d_group_ptr, d_sorted_idx);

  dh::XGBDeviceAllocator<char> alloc;
  auto labels = info.labels.View(device);
  if (thrust::any_of(thrust::cuda::par(alloc), dh::tbegin(labels.Values()),
                     dh::tend(labels.Values()), PRAUCLabelInvalid{})) {
    InvalidLabels();
  }
  /**
   * Get total positive/negative for each group.
   */
  auto d_weights = info.weights_.ConstDeviceSpan();
  dh::caching_device_vector<thrust::pair<double, double>> totals(n_groups);
  auto key_it = dh::MakeTransformIterator<size_t>(
      thrust::make_counting_iterator(0ul),
      [=] XGBOOST_DEVICE(size_t i) { return dh::SegmentId(d_group_ptr, i); });
  auto val_it = dh::MakeTransformIterator<Pair>(
      thrust::make_counting_iterator(0ul), [=] XGBOOST_DEVICE(size_t i) {
        float w = 1.0f;
        if (!d_weights.empty()) {
          // Avoid a binary search if the groups are not weighted.
          auto g = dh::SegmentId(d_group_ptr, i);
          w = d_weights[g];
        }
        auto y = labels(i);
        return thrust::make_pair(y * w, (1.0 - y) * w);
      });
  thrust::reduce_by_key(thrust::cuda::par(alloc), key_it,
                        key_it + predts.size(), val_it,
                        thrust::make_discard_iterator(), totals.begin(),
                        thrust::equal_to<size_t>{}, PairPlus<double, double>{});

  /**
   * Calculate AUC
   */
  auto d_totals = dh::ToSpan(totals);
  auto fn = [d_totals] XGBOOST_DEVICE(double fp_prev, double fp, double tp_prev,
                                      double tp, size_t group_id) {
    auto total_pos = d_totals[group_id].first;
    return detail::CalcDeltaPRAUC(fp_prev, fp, tp_prev, tp,
                                  d_totals[group_id].first);
  };
  return GPURankingPRAUCImpl(predts, info, d_group_ptr, device, cache, fn);
}
}  // namespace metric
}  // namespace xgboost
