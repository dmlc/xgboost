#include "hip/hip_runtime.h"
/*!
 * Copyright 2015-2019 by Contributors
 * \file multiclass_metric.cc
 * \brief evaluation metrics for multiclass classification.
 * \author Kailong Chen, Tianqi Chen
 */
#include <rabit/rabit.h>
#include <xgboost/metric.h>

#include <atomic>
#include <cmath>

#include "metric_common.h"
#include "../common/math.h"
#include "../common/common.h"
#include "../common/threading_utils.h"

#if defined(XGBOOST_USE_CUDA)
#include <thrust/execution_policy.h>  // thrust::cuda::par
#include <thrust/functional.h>        // thrust::plus<>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>

#include "../common/device_helpers.cuh"
#endif  // XGBOOST_USE_CUDA

namespace xgboost {
namespace metric {
// tag the this file, used by force static link later.
DMLC_REGISTRY_FILE_TAG(multiclass_metric);

template <typename EvalRowPolicy>
class MultiClassMetricsReduction {
  void CheckLabelError(int32_t label_error, size_t n_class) const {
    CHECK(label_error >= 0 && label_error < static_cast<int32_t>(n_class))
        << "MultiClassEvaluation: label must be in [0, num_class),"
        << " num_class=" << n_class << " but found " << label_error << " in label";
  }

 public:
  MultiClassMetricsReduction() = default;

  PackedReduceResult
  CpuReduceMetrics(const HostDeviceVector<bst_float> &weights,
                   const HostDeviceVector<bst_float> &labels,
                   const HostDeviceVector<bst_float> &preds,
                   const size_t n_class, int32_t n_threads) const {
    size_t ndata = labels.Size();

    const auto& h_labels = labels.HostVector();
    const auto& h_weights = weights.HostVector();
    const auto& h_preds = preds.HostVector();

    std::atomic<int> label_error {0};
    bool const is_null_weight = weights.Size() == 0;

    std::vector<double> scores_tloc(n_threads, 0);
    std::vector<double> weights_tloc(n_threads, 0);
    common::ParallelFor(ndata, n_threads, [&](size_t idx) {
        bst_float weight = is_null_weight ? 1.0f : h_weights[idx];
        auto label = static_cast<int>(h_labels[idx]);
        if (label >= 0 && label < static_cast<int>(n_class)) {
          auto t_idx = omp_get_thread_num();
          scores_tloc[t_idx] +=
              EvalRowPolicy::EvalRow(label, h_preds.data() + idx * n_class,
                                     n_class) *
              weight;
          weights_tloc[t_idx] += weight;
        } else {
          label_error = label;
        }
    });

    double residue_sum =
        std::accumulate(scores_tloc.cbegin(), scores_tloc.cend(), 0.0);
    double weights_sum =
        std::accumulate(weights_tloc.cbegin(), weights_tloc.cend(), 0.0);

    CheckLabelError(label_error, n_class);
    PackedReduceResult res { residue_sum, weights_sum };

    return res;
  }

#if defined(XGBOOST_USE_CUDA)

  PackedReduceResult DeviceReduceMetrics(
      const HostDeviceVector<bst_float>& weights,
      const HostDeviceVector<bst_float>& labels,
      const HostDeviceVector<bst_float>& preds,
      const size_t n_class) {
    size_t n_data = labels.Size();

    thrust::counting_iterator<size_t> begin(0);
    thrust::counting_iterator<size_t> end = begin + n_data;

    auto s_labels = labels.DeviceSpan();
    auto s_preds = preds.DeviceSpan();
    auto s_weights = weights.DeviceSpan();

    bool const is_null_weight = weights.Size() == 0;
    auto s_label_error = label_error_.GetSpan<int32_t>(1);
    s_label_error[0] = 0;

    dh::XGBCachingDeviceAllocator<char> alloc;
    PackedReduceResult result = thrust::transform_reduce(
        thrust::cuda::par(alloc),
        begin, end,
        [=] XGBOOST_DEVICE(size_t idx) {
          bst_float weight = is_null_weight ? 1.0f : s_weights[idx];
          bst_float residue = 0;
          auto label = static_cast<int>(s_labels[idx]);
          if (label >= 0 && label < static_cast<int32_t>(n_class)) {
            residue = EvalRowPolicy::EvalRow(
                label, &s_preds[idx * n_class], n_class) * weight;
          } else {
            s_label_error[0] = label;
          }
          return PackedReduceResult{ residue, weight };
        },
        PackedReduceResult(),
        thrust::plus<PackedReduceResult>());
    CheckLabelError(s_label_error[0], n_class);

    return result;
  }

#endif  // XGBOOST_USE_CUDA

  PackedReduceResult Reduce(
      const GenericParameter &tparam,
      int device,
      size_t n_class,
      const HostDeviceVector<bst_float>& weights,
      const HostDeviceVector<bst_float>& labels,
      const HostDeviceVector<bst_float>& preds) {
    PackedReduceResult result;

    if (device < 0) {
      result =
          CpuReduceMetrics(weights, labels, preds, n_class, tparam.Threads());
    }
#if defined(XGBOOST_USE_CUDA)
    else {  // NOLINT
      device_ = tparam.gpu_id;
      preds.SetDevice(device_);
      labels.SetDevice(device_);
      weights.SetDevice(device_);

      dh::safe_cuda(hipSetDevice(device_));
      result = DeviceReduceMetrics(weights, labels, preds, n_class);
    }
#endif  // defined(XGBOOST_USE_CUDA)
    return result;
  }

 private:
#if defined(XGBOOST_USE_CUDA)
  dh::PinnedMemory label_error_;
  int device_{-1};
#endif  // defined(XGBOOST_USE_CUDA)
};

/*!
 * \brief base class of multi-class evaluation
 * \tparam Derived the name of subclass
 */
template<typename Derived>
struct EvalMClassBase : public Metric {
  double Eval(const HostDeviceVector<float> &preds, const MetaInfo &info) override {
    if (info.labels.Size() == 0) {
      CHECK_EQ(preds.Size(), 0);
    } else {
      CHECK(preds.Size() % info.labels.Size() == 0) << "label and prediction size not match";
    }
    double dat[2] { 0.0, 0.0 };
    if (info.labels.Size() != 0) {
      const size_t nclass = preds.Size() / info.labels.Size();
      CHECK_GE(nclass, 1U)
          << "mlogloss and merror are only used for multi-class classification,"
          << " use logloss for binary classification";
      int device = tparam_->gpu_id;
      auto result =
          reducer_.Reduce(*tparam_, device, nclass, info.weights_, *info.labels.Data(), preds);
      dat[0] = result.Residue();
      dat[1] = result.Weights();
    }
    rabit::Allreduce<rabit::op::Sum>(dat, 2);
    return Derived::GetFinal(dat[0], dat[1]);
  }
  /*!
   * \brief to be implemented by subclass,
   *   get evaluation result from one row
   * \param label label of current instance
   * \param pred prediction value of current instance
   * \param nclass number of class in the prediction
   */
  XGBOOST_DEVICE static bst_float EvalRow(int label,
                                          const bst_float *pred,
                                          size_t nclass);
  /*!
   * \brief to be overridden by subclass, final transformation
   * \param esum the sum statistics returned by EvalRow
   * \param wsum sum of weight
   */
  inline static double GetFinal(double esum, double wsum) {
    return esum / wsum;
  }

 private:
  MultiClassMetricsReduction<Derived> reducer_;
  // used to store error message
  const char *error_msg_;
};

/*! \brief match error */
struct EvalMatchError : public EvalMClassBase<EvalMatchError> {
  const char* Name() const override {
    return "merror";
  }
  XGBOOST_DEVICE static bst_float EvalRow(int label,
                                          const bst_float *pred,
                                          size_t nclass) {
    return common::FindMaxIndex(pred, pred + nclass) != pred + static_cast<int>(label);
  }
};

/*! \brief match error */
struct EvalMultiLogLoss : public EvalMClassBase<EvalMultiLogLoss> {
  const char* Name() const override {
    return "mlogloss";
  }
  XGBOOST_DEVICE static bst_float EvalRow(int label, const bst_float* pred, size_t /*nclass*/) {
    const bst_float eps = 1e-16f;
    auto k = static_cast<size_t>(label);
    if (pred[k] > eps) {
      return -std::log(pred[k]);
    } else {
      return -std::log(eps);
    }
  }
};

XGBOOST_REGISTER_METRIC(MatchError, "merror")
    .describe("Multiclass classification error.")
    .set_body([](const char*) { return new EvalMatchError(); });

XGBOOST_REGISTER_METRIC(MultiLogLoss, "mlogloss")
    .describe("Multiclass negative loglikelihood.")
    .set_body([](const char*) { return new EvalMultiLogLoss(); });
}  // namespace metric
}  // namespace xgboost
