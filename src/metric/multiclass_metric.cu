#include "hip/hip_runtime.h"
/*!
 * Copyright 2015-2019 by Contributors
 * \file multiclass_metric.cc
 * \brief evaluation metrics for multiclass classification.
 * \author Kailong Chen, Tianqi Chen
 */
#include <rabit/rabit.h>
#include <xgboost/metric.h>
#include <cmath>

#include "metric_common.h"
#include "../common/math.h"
#include "../common/common.h"

#if defined(XGBOOST_USE_CUDA)
#include <thrust/execution_policy.h>  // thrust::cuda::par
#include <thrust/functional.h>        // thrust::plus<>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>

#include "../common/device_helpers.cuh"
#endif  // XGBOOST_USE_CUDA

namespace xgboost {
namespace metric {
// tag the this file, used by force static link later.
DMLC_REGISTRY_FILE_TAG(multiclass_metric);

template <typename EvalRowPolicy>
class MultiClassMetricsReduction {
  void CheckLabelError(int32_t label_error, size_t n_class) const {
    CHECK(label_error >= 0 && label_error < static_cast<int32_t>(n_class))
        << "MultiClassEvaluation: label must be in [0, num_class),"
        << " num_class=" << n_class << " but found " << label_error << " in label";
  }

 public:
  MultiClassMetricsReduction() = default;

  PackedReduceResult CpuReduceMetrics(
      const HostDeviceVector<bst_float>& weights,
      const HostDeviceVector<bst_float>& labels,
      const HostDeviceVector<bst_float>& preds,
      const size_t n_class) const {
    size_t ndata = labels.Size();

    const auto& h_labels = labels.HostVector();
    const auto& h_weights = weights.HostVector();
    const auto& h_preds = preds.HostVector();

    bst_float residue_sum = 0;
    bst_float weights_sum = 0;
    int label_error = 0;
    bool const is_null_weight = weights.Size() == 0;

#pragma omp parallel for reduction(+: residue_sum, weights_sum) schedule(static)
    for (omp_ulong idx = 0; idx < ndata; ++idx) {
      bst_float weight = is_null_weight ? 1.0f : h_weights[idx];
      auto label = static_cast<int>(h_labels[idx]);
      if (label >= 0 && label < static_cast<int>(n_class)) {
        residue_sum += EvalRowPolicy::EvalRow(
            label, h_preds.data() + idx * n_class, n_class) * weight;
        weights_sum += weight;
      } else {
        label_error = label;
      }
    }
    CheckLabelError(label_error, n_class);
    PackedReduceResult res { residue_sum, weights_sum };

    return res;
  }

#if defined(XGBOOST_USE_CUDA)

  PackedReduceResult DeviceReduceMetrics(
      GPUSet::GpuIdType device_id,
      size_t device_index,
      const HostDeviceVector<bst_float>& weights,
      const HostDeviceVector<bst_float>& labels,
      const HostDeviceVector<bst_float>& preds,
      const size_t n_class) {
    size_t n_data = labels.DeviceSize(device_id);

    thrust::counting_iterator<size_t> begin(0);
    thrust::counting_iterator<size_t> end = begin + n_data;

    auto s_labels = labels.DeviceSpan(device_id);
    auto s_preds = preds.DeviceSpan(device_id);
    auto s_weights = weights.DeviceSpan(device_id);

    bool const is_null_weight = weights.Size() == 0;
    auto s_label_error = label_error_.GetSpan<int32_t>(1);
    s_label_error[0] = 0;

    PackedReduceResult result = thrust::transform_reduce(
        thrust::cuda::par(allocators_.at(device_index)),
        begin, end,
        [=] XGBOOST_DEVICE(size_t idx) {
          bst_float weight = is_null_weight ? 1.0f : s_weights[idx];
          bst_float residue = 0;
          auto label = static_cast<int>(s_labels[idx]);
          if (label >= 0 && label < static_cast<int32_t>(n_class)) {
            residue = EvalRowPolicy::EvalRow(
                label, &s_preds[idx * n_class], n_class) * weight;
          } else {
            s_label_error[0] = label;
          }
          return PackedReduceResult{ residue, weight };
        },
        PackedReduceResult(),
        thrust::plus<PackedReduceResult>());
    CheckLabelError(s_label_error[0], n_class);

    return result;
  }

#endif  // XGBOOST_USE_CUDA

  PackedReduceResult Reduce(
      GPUSet devices,
      size_t n_class,
      const HostDeviceVector<bst_float>& weights,
      const HostDeviceVector<bst_float>& labels,
      const HostDeviceVector<bst_float>& preds) {
    PackedReduceResult result;

    if (devices.IsEmpty()) {
      result = CpuReduceMetrics(weights, labels, preds, n_class);
    }
#if defined(XGBOOST_USE_CUDA)
    else {  // NOLINT
      if (allocators_.size() != devices.Size()) {
        allocators_.clear();
        allocators_.resize(devices.Size());
      }
      preds.Shard(GPUDistribution::Granular(devices, n_class));
      labels.Shard(devices);
      weights.Shard(devices);
      std::vector<PackedReduceResult> res_per_device(devices.Size());

#pragma omp parallel for schedule(static, 1) if (devices.Size() > 1)
      for (GPUSet::GpuIdType id = *devices.begin(); id < *devices.end(); ++id) {
        dh::safe_cuda(hipSetDevice(id));
        size_t index = devices.Index(id);
        res_per_device.at(index) =
            DeviceReduceMetrics(id, index, weights, labels, preds, n_class);
      }

      for (auto const& res : res_per_device) {
        result += res;
      }
    }
#endif  // defined(XGBOOST_USE_CUDA)
    return result;
  }

 private:
#if defined(XGBOOST_USE_CUDA)
  dh::PinnedMemory label_error_;
  std::vector<dh::CubMemory> allocators_;
#endif  // defined(XGBOOST_USE_CUDA)
};

/*!
 * \brief base class of multi-class evaluation
 * \tparam Derived the name of subclass
 */
template<typename Derived>
struct EvalMClassBase : public Metric {
  bst_float Eval(const HostDeviceVector<bst_float> &preds,
                 const MetaInfo &info,
                 bool distributed) override {
    CHECK_NE(info.labels_.Size(), 0U) << "label set cannot be empty";
    CHECK(preds.Size() % info.labels_.Size() == 0)
        << "label and prediction size not match";
    const size_t nclass = preds.Size() / info.labels_.Size();
    CHECK_GE(nclass, 1U)
        << "mlogloss and merror are only used for multi-class classification,"
        << " use logloss for binary classification";
    const auto ndata = static_cast<bst_omp_uint>(info.labels_.Size());

    GPUSet devices = GPUSet::All(tparam_->gpu_id, tparam_->n_gpus, ndata);
    auto result = reducer_.Reduce(devices, nclass, info.weights_, info.labels_, preds);
    double dat[2] { result.Residue(), result.Weights() };

    if (distributed) {
      rabit::Allreduce<rabit::op::Sum>(dat, 2);
    }
    return Derived::GetFinal(dat[0], dat[1]);
  }
  /*!
   * \brief to be implemented by subclass,
   *   get evaluation result from one row
   * \param label label of current instance
   * \param pred prediction value of current instance
   * \param nclass number of class in the prediction
   */
  XGBOOST_DEVICE static bst_float EvalRow(int label,
                                          const bst_float *pred,
                                          size_t nclass);
  /*!
   * \brief to be overridden by subclass, final transformation
   * \param esum the sum statistics returned by EvalRow
   * \param wsum sum of weight
   */
  inline static bst_float GetFinal(bst_float esum, bst_float wsum) {
    return esum / wsum;
  }

 private:
  MultiClassMetricsReduction<Derived> reducer_;
  // used to store error message
  const char *error_msg_;
};

/*! \brief match error */
struct EvalMatchError : public EvalMClassBase<EvalMatchError> {
  const char* Name() const override {
    return "merror";
  }
  XGBOOST_DEVICE static bst_float EvalRow(int label,
                                          const bst_float *pred,
                                          size_t nclass) {
    return common::FindMaxIndex(pred, pred + nclass) != pred + static_cast<int>(label);
  }
};

/*! \brief match error */
struct EvalMultiLogLoss : public EvalMClassBase<EvalMultiLogLoss> {
  const char* Name() const override {
    return "mlogloss";
  }
  XGBOOST_DEVICE static bst_float EvalRow(int label,
                                          const bst_float *pred,
                                          size_t nclass) {
    const bst_float eps = 1e-16f;
    auto k = static_cast<size_t>(label);
    if (pred[k] > eps) {
      return -std::log(pred[k]);
    } else {
      return -std::log(eps);
    }
  }
};

XGBOOST_REGISTER_METRIC(MatchError, "merror")
.describe("Multiclass classification error.")
.set_body([](const char* param) { return new EvalMatchError(); });

XGBOOST_REGISTER_METRIC(MultiLogLoss, "mlogloss")
.describe("Multiclass negative loglikelihood.")
.set_body([](const char* param) { return new EvalMultiLogLoss(); });
}  // namespace metric
}  // namespace xgboost
