/*!
 * Copyright 2018-2019 by Contributors
 * \author Rory Mitchell
 */

#include <thrust/execution_policy.h>
#include <thrust/inner_product.h>
#include <xgboost/data.h>
#include <xgboost/linear_updater.h>
#include "xgboost/span.h"

#include "coordinate_common.h"
#include "../common/common.h"
#include "../common/device_helpers.cuh"
#include "../common/timer.h"
#include "./param.h"

namespace xgboost {
namespace linear {

DMLC_REGISTRY_FILE_TAG(updater_gpu_coordinate);

/**
 * \class GPUCoordinateUpdater
 *
 * \brief Coordinate descent algorithm that updates one feature per iteration
 */

class GPUCoordinateUpdater : public LinearUpdater {  // NOLINT
 public:
  // set training parameter
  void Configure(Args const& args) override {
    tparam_.UpdateAllowUnknown(args);
    coord_param_.UpdateAllowUnknown(args);
    selector_.reset(FeatureSelector::Create(tparam_.feature_selector, ctx_->Threads()));
    monitor_.Init("GPUCoordinateUpdater");
  }

  void LoadConfig(Json const& in) override {
    auto const& config = get<Object const>(in);
    FromJson(config.at("linear_train_param"), &tparam_);
    FromJson(config.at("coordinate_param"), &coord_param_);
  }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["linear_train_param"] = ToJson(tparam_);
    out["coordinate_param"] = ToJson(coord_param_);
  }

  void LazyInitDevice(DMatrix *p_fmat, const LearnerModelParam &model_param) {
    if (ctx_->gpu_id < 0) return;

    num_row_ = static_cast<size_t>(p_fmat->Info().num_row_);

    CHECK(p_fmat->SingleColBlock());
    SparsePage const& batch = *(p_fmat->GetBatches<CSCPage>().begin());
    auto page = batch.GetView();

    if (IsEmpty()) {
      return;
    }

    dh::safe_cuda(hipSetDevice(ctx_->gpu_id));
    // The begin and end indices for the section of each column associated with
    // this device
    std::vector<std::pair<bst_uint, bst_uint>> column_segments;
    row_ptr_ = {0};
    // iterate through columns
    for (size_t fidx = 0; fidx < batch.Size(); fidx++) {
      common::Span<Entry const> col = page[fidx];
      auto cmp = [](Entry e1, Entry e2) {
        return e1.index < e2.index;
      };
      auto column_begin =
          std::lower_bound(col.cbegin(), col.cend(),
                           xgboost::Entry(0, 0.0f), cmp);
      auto column_end =
          std::lower_bound(col.cbegin(), col.cend(),
                           xgboost::Entry(num_row_, 0.0f), cmp);
      column_segments.emplace_back(static_cast<bst_uint>(column_begin - col.cbegin()),
                                   static_cast<bst_uint>(column_end - col.cbegin()));
      row_ptr_.push_back(row_ptr_.back() + (column_end - column_begin));
    }
    data_.resize(row_ptr_.back());
    gpair_.resize(num_row_ * model_param.num_output_group);
    for (size_t fidx = 0; fidx < batch.Size(); fidx++) {
      auto col = page[fidx];
      auto seg = column_segments[fidx];
      dh::safe_cuda(hipMemcpy(
          data_.data().get() + row_ptr_[fidx],
          col.data() + seg.first,
          sizeof(Entry) * (seg.second - seg.first), hipMemcpyHostToDevice));
    }
  }

  void Update(HostDeviceVector<GradientPair> *in_gpair, DMatrix *p_fmat,
              gbm::GBLinearModel *model, double sum_instance_weight) override {
    tparam_.DenormalizePenalties(sum_instance_weight);
    monitor_.Start("LazyInitDevice");
    this->LazyInitDevice(p_fmat, *(model->learner_model_param));
    monitor_.Stop("LazyInitDevice");

    monitor_.Start("UpdateGpair");
    auto &in_gpair_host = in_gpair->ConstHostVector();
    // Update gpair
    if (ctx_->gpu_id >= 0) {
      this->UpdateGpair(in_gpair_host);
    }
    monitor_.Stop("UpdateGpair");

    monitor_.Start("UpdateBias");
    this->UpdateBias(model);
    monitor_.Stop("UpdateBias");
    // prepare for updating the weights
    selector_->Setup(*model, in_gpair->ConstHostVector(), p_fmat,
                     tparam_.reg_alpha_denorm, tparam_.reg_lambda_denorm,
                     coord_param_.top_k);
    monitor_.Start("UpdateFeature");
    for (uint32_t group_idx = 0; group_idx < model->learner_model_param->num_output_group;
         ++group_idx) {
      for (auto i = 0U; i < model->learner_model_param->num_feature; i++) {
        auto fidx = selector_->NextFeature(
            i, *model, group_idx, in_gpair->ConstHostVector(), p_fmat,
            tparam_.reg_alpha_denorm, tparam_.reg_lambda_denorm);
        if (fidx < 0) break;
        this->UpdateFeature(fidx, group_idx, model);
      }
    }
    monitor_.Stop("UpdateFeature");
  }

  void UpdateBias(gbm::GBLinearModel *model) {
    for (uint32_t group_idx = 0; group_idx < model->learner_model_param->num_output_group;
         ++group_idx) {
      // Get gradient
      auto grad = GradientPair(0, 0);
      if (ctx_->gpu_id >= 0) {
        grad = GetBiasGradient(group_idx, model->learner_model_param->num_output_group);
      }
      auto dbias = static_cast<float>(
          tparam_.learning_rate *
              CoordinateDeltaBias(grad.GetGrad(), grad.GetHess()));
      model->Bias()[group_idx] += dbias;

      // Update residual
      if (ctx_->gpu_id >= 0) {
        UpdateBiasResidual(dbias, group_idx, model->learner_model_param->num_output_group);
      }
    }
  }

  void UpdateFeature(int fidx, int group_idx,
                     gbm::GBLinearModel *model) {
    bst_float &w = (*model)[fidx][group_idx];
    // Get gradient
    auto grad = GradientPair(0, 0);
    if (ctx_->gpu_id >= 0) {
      grad = GetGradient(group_idx, model->learner_model_param->num_output_group, fidx);
    }
    auto dw = static_cast<float>(tparam_.learning_rate *
                                 CoordinateDelta(grad.GetGrad(), grad.GetHess(),
                                                 w, tparam_.reg_alpha_denorm,
                                                 tparam_.reg_lambda_denorm));
    w += dw;

    if (ctx_->gpu_id >= 0) {
      UpdateResidual(dw, group_idx, model->learner_model_param->num_output_group, fidx);
    }
  }

  // This needs to be public because of the __device__ lambda.
  GradientPair GetBiasGradient(int group_idx, int num_group) {
    dh::safe_cuda(hipSetDevice(ctx_->gpu_id));
    auto counting = thrust::make_counting_iterator(0ull);
    auto f = [=] __device__(size_t idx) {
      return idx * num_group + group_idx;
    };  // NOLINT
    thrust::transform_iterator<decltype(f), decltype(counting), size_t> skip(
        counting, f);
    auto perm = thrust::make_permutation_iterator(gpair_.data(), skip);

    return dh::SumReduction(perm, num_row_);
  }

  // This needs to be public because of the __device__ lambda.
  void UpdateBiasResidual(float dbias, int group_idx, int num_groups) {
    if (dbias == 0.0f) return;
    auto d_gpair = dh::ToSpan(gpair_);
    dh::LaunchN(num_row_, [=] __device__(size_t idx) {
      auto &g = d_gpair[idx * num_groups + group_idx];
      g += GradientPair(g.GetHess() * dbias, 0);
    });
  }

  // This needs to be public because of the __device__ lambda.
  GradientPair GetGradient(int group_idx, int num_group, int fidx) {
    dh::safe_cuda(hipSetDevice(ctx_->gpu_id));
    common::Span<xgboost::Entry> d_col = dh::ToSpan(data_).subspan(row_ptr_[fidx]);
    size_t col_size = row_ptr_[fidx + 1] - row_ptr_[fidx];
    common::Span<GradientPair> d_gpair = dh::ToSpan(gpair_);
    auto counting = thrust::make_counting_iterator(0ull);
    auto f = [=] __device__(size_t idx) {
      auto entry = d_col[idx];
      auto g = d_gpair[entry.index * num_group + group_idx];
      return GradientPair(g.GetGrad() * entry.fvalue,
                          g.GetHess() * entry.fvalue * entry.fvalue);
    };  // NOLINT
    thrust::transform_iterator<decltype(f), decltype(counting), GradientPair>
        multiply_iterator(counting, f);
    return dh::SumReduction(multiply_iterator, col_size);
  }

  // This needs to be public because of the __device__ lambda.
  void UpdateResidual(float dw, int group_idx, int num_groups, int fidx) {
    common::Span<GradientPair> d_gpair = dh::ToSpan(gpair_);
    common::Span<Entry> d_col = dh::ToSpan(data_).subspan(row_ptr_[fidx]);
    size_t col_size = row_ptr_[fidx + 1] - row_ptr_[fidx];
    dh::LaunchN(col_size, [=] __device__(size_t idx) {
      auto entry = d_col[idx];
      auto &g = d_gpair[entry.index * num_groups + group_idx];
      g += GradientPair(g.GetHess() * dw * entry.fvalue, 0);
    });
  }

 private:
  bool IsEmpty() {
    return num_row_ == 0;
  }

  void UpdateGpair(const std::vector<GradientPair> &host_gpair) {
    dh::safe_cuda(hipMemcpyAsync(
        gpair_.data().get(),
        host_gpair.data(),
        gpair_.size() * sizeof(GradientPair), hipMemcpyHostToDevice));
  }

  // training parameter
  LinearTrainParam tparam_;
  CoordinateParam coord_param_;
  std::unique_ptr<FeatureSelector> selector_;
  common::Monitor monitor_;

  std::vector<size_t> row_ptr_;
  dh::device_vector<xgboost::Entry> data_;
  dh::caching_device_vector<GradientPair> gpair_;
  size_t num_row_;
};

XGBOOST_REGISTER_LINEAR_UPDATER(GPUCoordinateUpdater, "gpu_coord_descent")
    .describe(
        "Update linear model according to coordinate descent algorithm. GPU "
        "accelerated.")
    .set_body([]() { return new GPUCoordinateUpdater(); });
}  // namespace linear
}  // namespace xgboost
