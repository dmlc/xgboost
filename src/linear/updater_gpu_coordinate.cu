#include "hip/hip_runtime.h"
/*!
 * Copyright 2018 by Contributors
 * \author Rory Mitchell
 */

#include <thrust/execution_policy.h>
#include <thrust/inner_product.h>
#include <xgboost/linear_updater.h>
#include "../common/device_helpers.cuh"
#include "../common/timer.h"
#include "coordinate_common.h"

namespace xgboost {
namespace linear {

DMLC_REGISTRY_FILE_TAG(updater_gpu_coordinate);

// training parameter
struct GPUCoordinateTrainParam
    : public dmlc::Parameter<GPUCoordinateTrainParam> {
  /*! \brief learning_rate */
  float learning_rate;
  /*! \brief regularization weight for L2 norm */
  float reg_lambda;
  /*! \brief regularization weight for L1 norm */
  float reg_alpha;
  int feature_selector;
  int top_k;
  int debug_verbose;
  int n_gpus;
  int gpu_id;
  bool silent;
  // declare parameters
  DMLC_DECLARE_PARAMETER(GPUCoordinateTrainParam) {
    DMLC_DECLARE_FIELD(learning_rate)
        .set_lower_bound(0.0f)
        .set_default(1.0f)
        .describe("Learning rate of each update.");
    DMLC_DECLARE_FIELD(reg_lambda)
        .set_lower_bound(0.0f)
        .set_default(0.0f)
        .describe("L2 regularization on weights.");
    DMLC_DECLARE_FIELD(reg_alpha)
        .set_lower_bound(0.0f)
        .set_default(0.0f)
        .describe("L1 regularization on weights.");
    DMLC_DECLARE_FIELD(feature_selector)
        .set_default(kCyclic)
        .add_enum("cyclic", kCyclic)
        .add_enum("shuffle", kShuffle)
        .add_enum("thrifty", kThrifty)
        .add_enum("greedy", kGreedy)
        .add_enum("random", kRandom)
        .describe("Feature selection or ordering method.");
    DMLC_DECLARE_FIELD(top_k).set_lower_bound(0).set_default(0).describe(
        "The number of top features to select in 'thrifty' feature_selector. "
        "The value of zero means using all the features.");
    DMLC_DECLARE_FIELD(debug_verbose)
        .set_lower_bound(0)
        .set_default(0)
        .describe("flag to print out detailed breakdown of runtime");
    DMLC_DECLARE_FIELD(n_gpus).set_default(1).describe(
        "Number of devices to use.");
    DMLC_DECLARE_FIELD(gpu_id).set_default(0).describe(
        "Primary device ordinal.");
    DMLC_DECLARE_FIELD(silent).set_default(false).describe(
        "Do not print information during trainig.");
    // alias of parameters
    DMLC_DECLARE_ALIAS(learning_rate, eta);
    DMLC_DECLARE_ALIAS(reg_lambda, lambda);
    DMLC_DECLARE_ALIAS(reg_alpha, alpha);
  }
  /*! \brief Denormalizes the regularization penalties - to be called at each
   * update */
  void DenormalizePenalties(double sum_instance_weight) {
    reg_lambda_denorm = reg_lambda * sum_instance_weight;
    reg_alpha_denorm = reg_alpha * sum_instance_weight;
  }
  // denormalizated regularization penalties
  float reg_lambda_denorm;
  float reg_alpha_denorm;
};

void RescaleIndices(size_t ridx_begin, dh::DVec<Entry> *data) {
  auto d_data = data->Data();
  dh::LaunchN(data->DeviceIdx(), data->Size(),
              [=] __device__(size_t idx) { d_data[idx].index -= ridx_begin; });
}

class DeviceShard {
  int device_idx_;
  int normalised_device_idx_;  // Device index counting from param.gpu_id
  dh::BulkAllocator<dh::MemoryType::kDevice> ba_;
  std::vector<size_t> row_ptr_;
  dh::DVec<Entry> data_;
  dh::DVec<GradientPair> gpair_;
  dh::CubMemory temp_;
  size_t ridx_begin_;
  size_t ridx_end_;

 public:
  DeviceShard(int device_idx, int normalised_device_idx, const SparsePage &batch,
              bst_uint row_begin, bst_uint row_end,
              const GPUCoordinateTrainParam &param,
              const gbm::GBLinearModelParam &model_param)
      : device_idx_(device_idx),
        normalised_device_idx_(normalised_device_idx),
        ridx_begin_(row_begin),
        ridx_end_(row_end) {
    dh::safe_cuda(hipSetDevice(device_idx));
    // The begin and end indices for the section of each column associated with
    // this shard
    std::vector<std::pair<bst_uint, bst_uint>> column_segments;
    row_ptr_ = {0};
    for (auto fidx = 0; fidx < batch.Size(); fidx++) {
      auto col = batch[fidx];
      auto cmp = [](Entry e1, Entry e2) {
        return e1.index < e2.index;
      };
      auto column_begin =
          std::lower_bound(col.data, col.data + col.length,
                           Entry(row_begin, 0.0f), cmp);
      auto column_end =
          std::upper_bound(col.data, col.data + col.length,
                           Entry(row_end, 0.0f), cmp);
      column_segments.push_back(
          std::make_pair(column_begin - col.data, column_end - col.data));
      row_ptr_.push_back(row_ptr_.back() + column_end - column_begin);
    }
    ba_.Allocate(device_idx, param.silent, &data_, row_ptr_.back(), &gpair_,
                (row_end - row_begin) * model_param.num_output_group);

    for (int fidx = 0; fidx < batch.Size(); fidx++) {
      auto col = batch[fidx];
      auto seg = column_segments[fidx];
      dh::safe_cuda(hipMemcpy(
          data_.Data() + row_ptr_[fidx], col.data + seg.first,
          sizeof(Entry) * (seg.second - seg.first), hipMemcpyHostToDevice));
    }
    // Rescale indices with respect to current shard
    RescaleIndices(ridx_begin_, &data_);
  }
  void UpdateGpair(const std::vector<GradientPair> &host_gpair,
                   const gbm::GBLinearModelParam &model_param) {
    gpair_.copy(host_gpair.begin() + ridx_begin_ * model_param.num_output_group,
               host_gpair.begin() + ridx_end_ * model_param.num_output_group);
  }

  GradientPair GetBiasGradient(int group_idx, int num_group) {
    auto counting = thrust::make_counting_iterator(0ull);
    auto f = [=] __device__(size_t idx) {
      return idx * num_group + group_idx;
    };  // NOLINT
    thrust::transform_iterator<decltype(f), decltype(counting), size_t> skip(
        counting, f);
    auto perm = thrust::make_permutation_iterator(gpair_.tbegin(), skip);

    return dh::SumReduction(temp_, perm, ridx_end_ - ridx_begin_);
  }

  void UpdateBiasResidual(float dbias, int group_idx, int num_groups) {
    if (dbias == 0.0f) return;
    auto d_gpair = gpair_.Data();
    dh::LaunchN(device_idx_, ridx_end_ - ridx_begin_, [=] __device__(size_t idx) {
      auto &g = d_gpair[idx * num_groups + group_idx];
      g += GradientPair(g.GetHess() * dbias, 0);
    });
  }

  GradientPair GetGradient(int group_idx, int num_group, int fidx) {
    auto d_col = data_.Data() + row_ptr_[fidx];
    size_t col_size = row_ptr_[fidx + 1] - row_ptr_[fidx];
    auto d_gpair = gpair_.Data();
    auto counting = thrust::make_counting_iterator(0ull);
    auto f = [=] __device__(size_t idx) {
      auto entry = d_col[idx];
      auto g = d_gpair[entry.index * num_group + group_idx];
      return GradientPair(g.GetGrad() * entry.fvalue,
                          g.GetHess() * entry.fvalue * entry.fvalue);
    };  // NOLINT
    thrust::transform_iterator<decltype(f), decltype(counting), GradientPair>
        multiply_iterator(counting, f);
    return dh::SumReduction(temp_, multiply_iterator, col_size);
  }

  void UpdateResidual(float dw, int group_idx, int num_groups, int fidx) {
    auto d_gpair = gpair_.Data();
    auto d_col = data_.Data() + row_ptr_[fidx];
    size_t col_size = row_ptr_[fidx + 1] - row_ptr_[fidx];
    dh::LaunchN(device_idx_, col_size, [=] __device__(size_t idx) {
      auto entry = d_col[idx];
      auto &g = d_gpair[entry.index * num_groups + group_idx];
      g += GradientPair(g.GetHess() * dw * entry.fvalue, 0);
    });
  }
};

/**
 * \class GPUCoordinateUpdater
 *
 * \brief Coordinate descent algorithm that updates one feature per iteration
 */

class GPUCoordinateUpdater : public LinearUpdater {
 public:
  // set training parameter
  void Init(
      const std::vector<std::pair<std::string, std::string>> &args) override {
    param.InitAllowUnknown(args);
    selector.reset(FeatureSelector::Create(param.feature_selector));
    monitor.Init("GPUCoordinateUpdater", param.debug_verbose);
  }

  void LazyInitShards(DMatrix *p_fmat,
                      const gbm::GBLinearModelParam &model_param) {
    if (!shards.empty()) return;
    int n_devices = dh::NDevices(param.n_gpus, p_fmat->Info().num_row_);
    bst_uint row_begin = 0;
    bst_uint shard_size =
        std::ceil(static_cast<double>(p_fmat->Info().num_row_) / n_devices);

    device_list.resize(n_devices);
    for (int d_idx = 0; d_idx < n_devices; ++d_idx) {
      int device_idx = (param.gpu_id + d_idx) % dh::NVisibleDevices();
      device_list[d_idx] = device_idx;
    }
    // Partition input matrix into row segments
    std::vector<size_t> row_segments;
    row_segments.push_back(0);
    for (int d_idx = 0; d_idx < n_devices; ++d_idx) {
      bst_uint row_end = std::min(static_cast<size_t>(row_begin + shard_size),
                                  p_fmat->Info().num_row_);
      row_segments.push_back(row_end);
      row_begin = row_end;
    }

    auto iter = p_fmat->ColIterator();
    CHECK(p_fmat->SingleColBlock());
    iter->Next();
    auto &batch = iter->Value();

    shards.resize(n_devices);
    // Create device shards
    dh::ExecuteShards(&shards, [&](std::unique_ptr<DeviceShard> &shard) {
      auto idx = &shard - &shards[0];
      shard = std::unique_ptr<DeviceShard>(
          new DeviceShard(device_list[idx], idx, batch, row_segments[idx],
                          row_segments[idx + 1], param, model_param));
    });
  }
  void Update(HostDeviceVector<GradientPair> *in_gpair, DMatrix *p_fmat,
              gbm::GBLinearModel *model, double sum_instance_weight) override {
    param.DenormalizePenalties(sum_instance_weight);
    monitor.Start("LazyInitShards");
    this->LazyInitShards(p_fmat, model->param);
    monitor.Stop("LazyInitShards");

    monitor.Start("UpdateGpair");
    // Update gpair
    dh::ExecuteShards(&shards, [&](std::unique_ptr<DeviceShard> &shard) {
      shard->UpdateGpair(in_gpair->HostVector(), model->param);
    });
    monitor.Stop("UpdateGpair");

    monitor.Start("UpdateBias");
    this->UpdateBias(p_fmat, model);
    monitor.Stop("UpdateBias");
    // prepare for updating the weights
    selector->Setup(*model, in_gpair->HostVector(), p_fmat,
                    param.reg_alpha_denorm, param.reg_lambda_denorm,
                    param.top_k);
    monitor.Start("UpdateFeature");
    for (auto group_idx = 0; group_idx < model->param.num_output_group;
         ++group_idx) {
      for (auto i = 0U; i < model->param.num_feature; i++) {
        auto fidx = selector->NextFeature(
            i, *model, group_idx, in_gpair->HostVector(), p_fmat,
            param.reg_alpha_denorm, param.reg_lambda_denorm);
        if (fidx < 0) break;
        this->UpdateFeature(fidx, group_idx, &in_gpair->HostVector(), model);
      }
    }
    monitor.Stop("UpdateFeature");
  }

  void UpdateBias(DMatrix *p_fmat, gbm::GBLinearModel *model) {
    for (int group_idx = 0; group_idx < model->param.num_output_group;
         ++group_idx) {
      // Get gradient
      auto grad = dh::ReduceShards<GradientPair>(
          &shards, [&](std::unique_ptr<DeviceShard> &shard) {
            return shard->GetBiasGradient(group_idx,
                                          model->param.num_output_group);
          });

      auto dbias = static_cast<float>(
          param.learning_rate *
          CoordinateDeltaBias(grad.GetGrad(), grad.GetHess()));
      model->bias()[group_idx] += dbias;

      // Update residual
      dh::ExecuteShards(&shards, [&](std::unique_ptr<DeviceShard> &shard) {
        shard->UpdateBiasResidual(dbias, group_idx,
                                  model->param.num_output_group);
      });
    }
  }

  void UpdateFeature(int fidx, int group_idx,
                     std::vector<GradientPair> *in_gpair,
                     gbm::GBLinearModel *model) {
    bst_float &w = (*model)[fidx][group_idx];
    // Get gradient
    auto grad = dh::ReduceShards<GradientPair>(
        &shards, [&](std::unique_ptr<DeviceShard> &shard) {
          return shard->GetGradient(group_idx, model->param.num_output_group,
                                    fidx);
        });

    auto dw = static_cast<float>(param.learning_rate *
                                 CoordinateDelta(grad.GetGrad(), grad.GetHess(),
                                                 w, param.reg_alpha_denorm,
                                                 param.reg_lambda_denorm));
    w += dw;

    dh::ExecuteShards(&shards, [&](std::unique_ptr<DeviceShard> &shard) {
      shard->UpdateResidual(dw, group_idx, model->param.num_output_group, fidx);
    });
  }

  // training parameter
  GPUCoordinateTrainParam param;
  std::unique_ptr<FeatureSelector> selector;
  common::Monitor monitor;

  std::vector<std::unique_ptr<DeviceShard>> shards;
  std::vector<int> device_list;
};

DMLC_REGISTER_PARAMETER(GPUCoordinateTrainParam);
XGBOOST_REGISTER_LINEAR_UPDATER(GPUCoordinateUpdater, "gpu_coord_descent")
    .describe(
        "Update linear model according to coordinate descent algorithm. GPU "
        "accelerated.")
    .set_body([]() { return new GPUCoordinateUpdater(); });
}  // namespace linear
}  // namespace xgboost
