#include "hip/hip_runtime.h"
/**
 * Copyright 2019-2025, XGBoost Contributors
 * \file aft_obj.cu
 * \brief Definition of AFT loss for survival analysis.
 * \author Avinash Barnwal, Hyunsu Cho and Toby Hocking
 */

#include <cmath>    // for log
#include <cstddef>  // for size_t


#include "../common/survival_util.h"
#include "../common/transform.h"
#include "xgboost/host_device_vector.h"
#include "xgboost/json.h"
#include "xgboost/logging.h"
#include "xgboost/objective.h"
#include "xgboost/span.h"

#if defined(XGBOOST_USE_CUDA)
#include "../common/linalg_op.cuh"  // for ElementWiseKernel
#elif defined(XGBOOST_USE_SYCL)
#include "../../plugin/sycl/common/linalg_op.h"
#else
#include "../common/linalg_op.h"  // for ElementWiseKernel
#endif

using AFTParam = xgboost::common::AFTParam;
using ProbabilityDistributionType = xgboost::common::ProbabilityDistributionType;
template <typename Distribution>
using AFTLoss = xgboost::common::AFTLoss<Distribution>;

namespace xgboost {
namespace obj {

#if defined(XGBOOST_USE_CUDA)
DMLC_REGISTRY_FILE_TAG(aft_obj_gpu);
#endif  // defined(XGBOOST_USE_CUDA)

class AFTObj : public ObjFunction {
 public:
  void Configure(Args const& args) override {
    param_.UpdateAllowUnknown(args);
  }

  ObjInfo Task() const override { return ObjInfo::kSurvival; }

  template <typename Distribution>
  void GetGradientImpl(const HostDeviceVector<bst_float>& preds, const MetaInfo& info,
                       linalg::Matrix<GradientPair>* out_gpair, size_t ndata, DeviceOrd device,
                       bool is_null_weight, float aft_loss_distribution_scale) {
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
        common::Span<GradientPair> _out_gpair,
        common::Span<const bst_float> _preds,
        common::Span<const bst_float> _labels_lower_bound,
        common::Span<const bst_float> _labels_upper_bound,
        common::Span<const bst_float> _weights) {
      const double pred = static_cast<double>(_preds[_idx]);
      const double label_lower_bound = static_cast<double>(_labels_lower_bound[_idx]);
      const double label_upper_bound = static_cast<double>(_labels_upper_bound[_idx]);
      const float grad = static_cast<float>(
          AFTLoss<Distribution>::Gradient(label_lower_bound, label_upper_bound,
                                          pred, aft_loss_distribution_scale));
      const float hess = static_cast<float>(
          AFTLoss<Distribution>::Hessian(label_lower_bound, label_upper_bound,
                                         pred, aft_loss_distribution_scale));
      const bst_float w = is_null_weight ? 1.0f : _weights[_idx];
      _out_gpair[_idx] = GradientPair(grad * w, hess * w);
    },
    common::Range{0, static_cast<int64_t>(ndata)}, this->ctx_->Threads(), device).Eval(
        out_gpair->Data(), &preds, &info.labels_lower_bound_, &info.labels_upper_bound_,
        &info.weights_);
  }

  void GetGradient(const HostDeviceVector<bst_float>& preds, const MetaInfo& info, int /*iter*/,
                   linalg::Matrix<GradientPair>* out_gpair) override {
    const size_t ndata = preds.Size();
    CHECK_EQ(info.labels_lower_bound_.Size(), ndata);
    CHECK_EQ(info.labels_upper_bound_.Size(), ndata);
    out_gpair->SetDevice(ctx_->Device());
    out_gpair->Reshape(ndata, 1);
    const auto device = ctx_->Device();
    const float aft_loss_distribution_scale = param_.aft_loss_distribution_scale;
    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
        << "Number of weights should be equal to number of data points.";
    }

    switch (param_.aft_loss_distribution) {
    case common::ProbabilityDistributionType::kNormal:
      GetGradientImpl<common::NormalDistribution>(preds, info, out_gpair, ndata, device,
                                                  is_null_weight, aft_loss_distribution_scale);
      break;
    case common::ProbabilityDistributionType::kLogistic:
      GetGradientImpl<common::LogisticDistribution>(preds, info, out_gpair, ndata, device,
                                                    is_null_weight, aft_loss_distribution_scale);
      break;
    case common::ProbabilityDistributionType::kExtreme:
      GetGradientImpl<common::ExtremeDistribution>(preds, info, out_gpair, ndata, device,
                                                   is_null_weight, aft_loss_distribution_scale);
      break;
    default:
      LOG(FATAL) << "Unrecognized distribution";
    }
  }

  void PredTransform(HostDeviceVector<bst_float> *io_preds) const override {
    // Trees give us a prediction in log scale, so exponentiate
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = exp(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())}, this->ctx_->Threads(),
        io_preds->Device())
        .Eval(io_preds);
  }

  void EvalTransform(HostDeviceVector<bst_float>* /*io_preds*/) override {
    // do nothing here, since the AFT metric expects untransformed prediction score
  }

  void ProbToMargin(linalg::Vector<float>* base_score) const override {
    auto intercept = base_score->View(this->ctx_->Device());
    linalg::ElementWiseKernel(ctx_, intercept, [=] XGBOOST_DEVICE(std::size_t i) mutable {
      intercept(i) = std::log(intercept(i));
    });
  }

  const char* DefaultEvalMetric() const override {
    return "aft-nloglik";
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("survival:aft");
    out["aft_loss_param"] = ToJson(param_);
  }

  void LoadConfig(Json const& in) override {
    FromJson(in["aft_loss_param"], &param_);
  }
  Json DefaultMetricConfig() const override {
    Json config{Object{}};
    config["name"] = String{this->DefaultEvalMetric()};
    config["aft_loss_param"] = ToJson(param_);
    return config;
  }

 private:
  AFTParam param_;
};

// register the objective functions
XGBOOST_REGISTER_OBJECTIVE(AFTObj, "survival:aft")
    .describe("AFT loss function")
    .set_body([]() { return new AFTObj(); });

}  // namespace obj
}  // namespace xgboost
