#include "hip/hip_runtime.h"
/*!
 * Copyright 2019-2022 by Contributors
 * \file aft_obj.cu
 * \brief Definition of AFT loss for survival analysis.
 * \author Avinash Barnwal, Hyunsu Cho and Toby Hocking
 */

#include <vector>
#include <limits>
#include <memory>
#include <utility>

#include "xgboost/host_device_vector.h"
#include "xgboost/json.h"
#include "xgboost/parameter.h"
#include "xgboost/span.h"
#include "xgboost/logging.h"
#include "xgboost/objective.h"

#include "../common/transform.h"
#include "../common/survival_util.h"

using AFTParam = xgboost::common::AFTParam;
using ProbabilityDistributionType = xgboost::common::ProbabilityDistributionType;
template <typename Distribution>
using AFTLoss = xgboost::common::AFTLoss<Distribution>;

namespace xgboost {
namespace obj {

#if defined(XGBOOST_USE_CUDA)
DMLC_REGISTRY_FILE_TAG(aft_obj_gpu);
#endif  // defined(XGBOOST_USE_CUDA)

class AFTObj : public ObjFunction {
 public:
  void Configure(Args const& args) override {
    param_.UpdateAllowUnknown(args);
  }

  ObjInfo Task() const override { return ObjInfo::kSurvival; }

  template <typename Distribution>
  void GetGradientImpl(const HostDeviceVector<bst_float> &preds,
                       const MetaInfo &info,
                       HostDeviceVector<GradientPair> *out_gpair,
                       size_t ndata, int device, bool is_null_weight,
                       float aft_loss_distribution_scale) {
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
        common::Span<GradientPair> _out_gpair,
        common::Span<const bst_float> _preds,
        common::Span<const bst_float> _labels_lower_bound,
        common::Span<const bst_float> _labels_upper_bound,
        common::Span<const bst_float> _weights) {
      const double pred = static_cast<double>(_preds[_idx]);
      const double label_lower_bound = static_cast<double>(_labels_lower_bound[_idx]);
      const double label_upper_bound = static_cast<double>(_labels_upper_bound[_idx]);
      const float grad = static_cast<float>(
          AFTLoss<Distribution>::Gradient(label_lower_bound, label_upper_bound,
                                          pred, aft_loss_distribution_scale));
      const float hess = static_cast<float>(
          AFTLoss<Distribution>::Hessian(label_lower_bound, label_upper_bound,
                                         pred, aft_loss_distribution_scale));
      const bst_float w = is_null_weight ? 1.0f : _weights[_idx];
      _out_gpair[_idx] = GradientPair(grad * w, hess * w);
    },
    common::Range{0, static_cast<int64_t>(ndata)}, this->ctx_->Threads(), device).Eval(
        out_gpair, &preds, &info.labels_lower_bound_, &info.labels_upper_bound_,
        &info.weights_);
  }

  void GetGradient(const HostDeviceVector<bst_float>& preds, const MetaInfo& info, int /*iter*/,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    const size_t ndata = preds.Size();
    CHECK_EQ(info.labels_lower_bound_.Size(), ndata);
    CHECK_EQ(info.labels_upper_bound_.Size(), ndata);
    out_gpair->Resize(ndata);
    const int device = ctx_->gpu_id;
    const float aft_loss_distribution_scale = param_.aft_loss_distribution_scale;
    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
        << "Number of weights should be equal to number of data points.";
    }

    switch (param_.aft_loss_distribution) {
    case common::ProbabilityDistributionType::kNormal:
      GetGradientImpl<common::NormalDistribution>(preds, info, out_gpair, ndata, device,
                                                  is_null_weight, aft_loss_distribution_scale);
      break;
    case common::ProbabilityDistributionType::kLogistic:
      GetGradientImpl<common::LogisticDistribution>(preds, info, out_gpair, ndata, device,
                                                    is_null_weight, aft_loss_distribution_scale);
      break;
    case common::ProbabilityDistributionType::kExtreme:
      GetGradientImpl<common::ExtremeDistribution>(preds, info, out_gpair, ndata, device,
                                                   is_null_weight, aft_loss_distribution_scale);
      break;
    default:
      LOG(FATAL) << "Unrecognized distribution";
    }
  }

  void PredTransform(HostDeviceVector<bst_float> *io_preds) const override {
    // Trees give us a prediction in log scale, so exponentiate
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = exp(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())}, this->ctx_->Threads(),
        io_preds->DeviceIdx())
        .Eval(io_preds);
  }

  void EvalTransform(HostDeviceVector<bst_float>* /*io_preds*/) override {
    // do nothing here, since the AFT metric expects untransformed prediction score
  }

  bst_float ProbToMargin(bst_float base_score) const override {
    return std::log(base_score);
  }

  const char* DefaultEvalMetric() const override {
    return "aft-nloglik";
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("survival:aft");
    out["aft_loss_param"] = ToJson(param_);
  }

  void LoadConfig(Json const& in) override {
    FromJson(in["aft_loss_param"], &param_);
  }

 private:
  AFTParam param_;
};

// register the objective functions
XGBOOST_REGISTER_OBJECTIVE(AFTObj, "survival:aft")
    .describe("AFT loss function")
    .set_body([]() { return new AFTObj(); });

}  // namespace obj
}  // namespace xgboost
