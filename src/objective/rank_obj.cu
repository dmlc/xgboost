#include "hip/hip_runtime.h"
/*!
 * Copyright 2015-2019 XGBoost contributors
 */
#include <dmlc/omp.h>
#include <dmlc/timer.h>
#include <xgboost/logging.h>
#include <xgboost/objective.h>
#include <vector>
#include <algorithm>
#include <utility>

#include "xgboost/json.h"
#include "xgboost/parameter.h"

#include "../common/math.h"
#include "../common/random.h"

#if defined(__HIPCC__)
#include <thrust/sort.h>
#include <thrust/gather.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/random/uniform_int_distribution.h>
#include <thrust/random/linear_congruential_engine.h>

#include <cub/util_allocator.cuh>

#include "../common/device_helpers.cuh"
#endif

namespace xgboost {
namespace obj {

#if defined(XGBOOST_USE_CUDA) && !defined(GTEST_TEST)
DMLC_REGISTRY_FILE_TAG(rank_obj_gpu);
#endif  // defined(XGBOOST_USE_CUDA)

struct LambdaRankParam : public XGBoostParameter<LambdaRankParam> {
  size_t num_pairsample;
  float fix_list_weight;
  // declare parameters
  DMLC_DECLARE_PARAMETER(LambdaRankParam) {
    DMLC_DECLARE_FIELD(num_pairsample).set_lower_bound(1).set_default(1)
        .describe("Number of pair generated for each instance.");
    DMLC_DECLARE_FIELD(fix_list_weight).set_lower_bound(0.0f).set_default(0.0f)
        .describe("Normalize the weight of each list by this value,"
                  " if equals 0, no effect will happen");
  }
};

#if defined(__HIPCC__)
// Helper functions

template <typename T>
XGBOOST_DEVICE __forceinline__ uint32_t
CountNumItemsToTheLeftOf(const T *__restrict__ items, uint32_t n, T v) {
  return thrust::lower_bound(thrust::seq, items, items + n, v,
                             thrust::greater<T>()) -
         items;
}

template <typename T>
XGBOOST_DEVICE __forceinline__ uint32_t
CountNumItemsToTheRightOf(const T *__restrict__ items, uint32_t n, T v) {
  return n - (thrust::upper_bound(thrust::seq, items, items + n, v,
                                  thrust::greater<T>()) -
              items);
}
#endif

/*! \brief helper information in a list */
struct ListEntry {
  /*! \brief the predict score we in the data */
  bst_float pred;
  /*! \brief the actual label of the entry */
  bst_float label;
  /*! \brief row index in the data matrix */
  unsigned rindex;
  // constructor
  ListEntry(bst_float pred, bst_float label, unsigned rindex)
    : pred(pred), label(label), rindex(rindex) {}
  // comparator by prediction
  inline static bool CmpPred(const ListEntry &a, const ListEntry &b) {
    return a.pred > b.pred;
  }
  // comparator by label
  inline static bool CmpLabel(const ListEntry &a, const ListEntry &b) {
    return a.label > b.label;
  }
};

/*! \brief a pair in the lambda rank */
struct LambdaPair {
  /*! \brief positive index: this is a position in the list */
  unsigned pos_index;
  /*! \brief negative index: this is a position in the list */
  unsigned neg_index;
  /*! \brief weight to be filled in */
  bst_float weight;
  // constructor
  LambdaPair(unsigned pos_index, unsigned neg_index)
    : pos_index(pos_index), neg_index(neg_index), weight(1.0f) {}
  // constructor
  LambdaPair(unsigned pos_index, unsigned neg_index, bst_float weight)
    : pos_index(pos_index), neg_index(neg_index), weight(weight) {}
};

class PairwiseLambdaWeightComputer {
 public:
  /*!
   * \brief get lambda weight for existing pairs - for pairwise objective
   * \param list a list that is sorted by pred score
   * \param io_pairs record of pairs, containing the pairs to fill in weights
   */
  static void GetLambdaWeight(const std::vector<ListEntry>&,
                              std::vector<LambdaPair>*) {}

  static char const* Name() {
    return "rank:pairwise";
  }

#if defined(__HIPCC__)
  PairwiseLambdaWeightComputer(const bst_float*,
                               const bst_float*,
                               const dh::SegmentSorter<float>&) {}

  class PairwiseLambdaWeightMultiplier {
   public:
    // Adjust the items weight by this value
    __device__ __forceinline__ bst_float GetWeight(uint32_t gidx, int pidx, int nidx) const {
      return 1.0f;
    }
  };

  inline const PairwiseLambdaWeightMultiplier GetWeightMultiplier() const {
    return {};
  }
#endif
};

#if defined(__HIPCC__)
class BaseLambdaWeightMultiplier {
 public:
  BaseLambdaWeightMultiplier(const dh::SegmentSorter<float> &segment_label_sorter,
                             const dh::SegmentSorter<float> &segment_pred_sorter)
    : dsorted_labels_(segment_label_sorter.GetItemsSpan()),
      dorig_pos_(segment_label_sorter.GetOriginalPositionsSpan()),
      dgroups_(segment_label_sorter.GetGroupsSpan()),
      dindexable_sorted_preds_pos_(segment_pred_sorter.GetIndexableSortedPositionsSpan()) {}

 protected:
  const common::Span<const float> dsorted_labels_;  // Labels sorted within a group
  const common::Span<const uint32_t> dorig_pos_;  // Original indices of the labels
                                                  // before they are sorted
  const common::Span<const uint32_t> dgroups_;  // The group indices
  // Where can a prediction for a label be found in the original array, when they are sorted
  const common::Span<const uint32_t> dindexable_sorted_preds_pos_;
};

// While computing the weight that needs to be adjusted by this ranking objective, we need
// to figure out where positive and negative labels chosen earlier exists, if the group
// were to be sorted by its predictions. To accommodate this, we employ the following algorithm.
// For a given group, let's assume the following:
// labels:        1 5 9 2 4 8 0 7 6 3
// predictions:   1 9 0 8 2 7 3 6 5 4
// position:      0 1 2 3 4 5 6 7 8 9
//
// After label sort:
// labels:        9 8 7 6 5 4 3 2 1 0
// position:      2 5 7 8 1 4 9 3 0 6
//
// After prediction sort:
// predictions:   9 8 7 6 5 4 3 2 1 0
// position:      1 3 5 7 8 9 6 4 0 2
//
// If a sorted label at position 'x' is chosen, then we need to find out where the prediction
// for this label 'x' exists, if the group were to be sorted by predictions.
// We first take the sorted prediction positions:
// position:      1 3 5 7 8 9 6 4 0 2
// at indices:    0 1 2 3 4 5 6 7 8 9
//
// We create a sorted prediction positional array, such that value at position 'x' gives
// us the position in the sorted prediction array where its related prediction lies.
// dindexable_sorted_preds_pos_:  8 0 9 1 7 2 6 3 4 5
// at indices:                    0 1 2 3 4 5 6 7 8 9
// Basically, swap the previous 2 arrays, sort the indices and reorder positions
// for an O(1) lookup using the position where the sorted label exists.
//
// This type does that using the SegmentSorter
class IndexablePredictionSorter {
 public:
  IndexablePredictionSorter(const bst_float *dpreds,
                            const dh::SegmentSorter<float> &segment_label_sorter) {
    // Sort the predictions first
    segment_pred_sorter_.SortItems(dpreds, segment_label_sorter.GetNumItems(),
                                   segment_label_sorter.GetGroupSegmentsSpan());

    // Create an index for the sorted prediction positions
    segment_pred_sorter_.CreateIndexableSortedPositions();
  }

  inline const dh::SegmentSorter<float> &GetPredictionSorter() const {
    return segment_pred_sorter_;
  }

 private:
  dh::SegmentSorter<float> segment_pred_sorter_;  // For sorting the predictions
};
#endif

// beta version: NDCG lambda rank
class NDCGLambdaWeightComputer
#if defined(__HIPCC__)
  : public IndexablePredictionSorter
#endif
{
 public:
#if defined(__HIPCC__)
  // This function object computes the item's DCG value
  class ComputeItemDCG : public thrust::unary_function<uint32_t, float> {
   public:
    XGBOOST_DEVICE ComputeItemDCG(const common::Span<const float> &dsorted_labels,
                                  const common::Span<const uint32_t> &dgroups,
                                  const common::Span<const uint32_t> &gidxs)
      : dsorted_labels_(dsorted_labels),
        dgroups_(dgroups),
        dgidxs_(gidxs) {}

    // Compute DCG for the item at 'idx'
    __device__ __forceinline__ float operator()(uint32_t idx) const {
      return ComputeItemDCGWeight(dsorted_labels_[idx], idx - dgroups_[dgidxs_[idx]]);
    }

   private:
    const common::Span<const float> dsorted_labels_;  // Labels sorted within a group
    const common::Span<const uint32_t> dgroups_;  // The group indices - where each group
                                                  // begins and ends
    const common::Span<const uint32_t> dgidxs_;  // The group each items belongs to
  };

  // Type containing device pointers that can be cheaply copied on the kernel
  class NDCGLambdaWeightMultiplier : public BaseLambdaWeightMultiplier {
   public:
    NDCGLambdaWeightMultiplier(const dh::SegmentSorter<float> &segment_label_sorter,
                               const NDCGLambdaWeightComputer &lwc)
      : BaseLambdaWeightMultiplier(segment_label_sorter, lwc.GetPredictionSorter()),
        dgroup_dcgs_(lwc.GetGroupDcgsSpan()) {}

    // Adjust the items weight by this value
    __device__ __forceinline__ bst_float GetWeight(uint32_t gidx, int pidx, int nidx) const {
      if (dgroup_dcgs_[gidx] == 0.0) return 0.0f;

      uint32_t group_begin = dgroups_[gidx];

      auto pos_lab_orig_posn = dorig_pos_[pidx];
      auto neg_lab_orig_posn = dorig_pos_[nidx];
      KERNEL_CHECK(pos_lab_orig_posn != neg_lab_orig_posn);

      // Note: the label positive and negative indices are relative to the entire dataset.
      // Hence, scale them back to an index within the group
      auto pos_pred_pos = dindexable_sorted_preds_pos_[pos_lab_orig_posn] - group_begin;
      auto neg_pred_pos = dindexable_sorted_preds_pos_[neg_lab_orig_posn] - group_begin;
      return NDCGLambdaWeightComputer::ComputeDeltaWeight(
        pos_pred_pos, neg_pred_pos,
        static_cast<int>(dsorted_labels_[pidx]), static_cast<int>(dsorted_labels_[nidx]),
        dgroup_dcgs_[gidx]);
    }

   private:
     const common::Span<const float> dgroup_dcgs_;  // Group DCG values
  };

  NDCGLambdaWeightComputer(const bst_float *dpreds,
                           const bst_float*,
                           const dh::SegmentSorter<float> &segment_label_sorter)
    : IndexablePredictionSorter(dpreds, segment_label_sorter),
      dgroup_dcg_(segment_label_sorter.GetNumGroups(), 0.0f),
      weight_multiplier_(segment_label_sorter, *this) {
    const auto &group_segments = segment_label_sorter.GetGroupSegmentsSpan();

    // Allocator to be used for managing space overhead while performing transformed reductions
    dh::XGBCachingDeviceAllocator<char> alloc;

    // Compute each elements DCG values and reduce them across groups concurrently.
    auto end_range =
      thrust::reduce_by_key(thrust::cuda::par(alloc),
                            dh::tcbegin(group_segments), dh::tcend(group_segments),
                            thrust::make_transform_iterator(
                              // The indices need not be sequential within a group, as we care only
                              // about the sum of items DCG values within a group
                              dh::tcbegin(segment_label_sorter.GetOriginalPositionsSpan()),
                              ComputeItemDCG(segment_label_sorter.GetItemsSpan(),
                                             segment_label_sorter.GetGroupsSpan(),
                                             group_segments)),
                            thrust::make_discard_iterator(),  // We don't care for the group indices
                            dgroup_dcg_.begin());  // Sum of the item's DCG values in the group
    CHECK(static_cast<unsigned>(end_range.second - dgroup_dcg_.begin()) == dgroup_dcg_.size());
  }

  inline const common::Span<const float> GetGroupDcgsSpan() const {
    return { dgroup_dcg_.data().get(), dgroup_dcg_.size() };
  }

  inline const NDCGLambdaWeightMultiplier GetWeightMultiplier() const {
    return weight_multiplier_;
  }
#endif

  static void GetLambdaWeight(const std::vector<ListEntry> &sorted_list,
                              std::vector<LambdaPair> *io_pairs) {
    std::vector<LambdaPair> &pairs = *io_pairs;
    float IDCG;  // NOLINT
    {
      std::vector<bst_float> labels(sorted_list.size());
      for (size_t i = 0; i < sorted_list.size(); ++i) {
        labels[i] = sorted_list[i].label;
      }
      std::stable_sort(labels.begin(), labels.end(), std::greater<>());
      IDCG = ComputeGroupDCGWeight(&labels[0], labels.size());
    }
    if (IDCG == 0.0) {
      for (auto & pair : pairs) {
        pair.weight = 0.0f;
      }
    } else {
      for (auto & pair : pairs) {
        unsigned pos_idx = pair.pos_index;
        unsigned neg_idx = pair.neg_index;
        pair.weight *= ComputeDeltaWeight(pos_idx, neg_idx,
                                          sorted_list[pos_idx].label, sorted_list[neg_idx].label,
                                          IDCG);
      }
    }
  }

  static char const* Name() {
    return "rank:ndcg";
  }

  inline static bst_float ComputeGroupDCGWeight(const float *sorted_labels, uint32_t size) {
    double sumdcg = 0.0;
    for (uint32_t i = 0; i < size; ++i) {
      sumdcg += ComputeItemDCGWeight(sorted_labels[i], i);
    }

    return static_cast<bst_float>(sumdcg);
  }

 private:
  XGBOOST_DEVICE inline static bst_float ComputeItemDCGWeight(unsigned label, uint32_t idx) {
    return (label != 0) ? (((1 << label) - 1) / std::log2(static_cast<bst_float>(idx + 2))) : 0;
  }

  // Compute the weight adjustment for an item within a group:
  // pos_pred_pos => Where does the positive label live, had the list been sorted by prediction
  // neg_pred_pos => Where does the negative label live, had the list been sorted by prediction
  // pos_label => positive label value from sorted label list
  // neg_label => negative label value from sorted label list
  XGBOOST_DEVICE inline static bst_float ComputeDeltaWeight(uint32_t pos_pred_pos,
                                                            uint32_t neg_pred_pos,
                                                            int pos_label, int neg_label,
                                                            float idcg) {
    float pos_loginv = 1.0f / std::log2(pos_pred_pos + 2.0f);
    float neg_loginv = 1.0f / std::log2(neg_pred_pos + 2.0f);
    bst_float original = ((1 << pos_label) - 1) * pos_loginv + ((1 << neg_label) - 1) * neg_loginv;
    float changed = ((1 << neg_label) - 1) * pos_loginv + ((1 << pos_label) - 1) * neg_loginv;
    bst_float delta = (original - changed) * (1.0f / idcg);
    if (delta < 0.0f) delta = - delta;
    return delta;
  }

#if defined(__HIPCC__)
  dh::caching_device_vector<float> dgroup_dcg_;
  // This computes the adjustment to the weight
  const NDCGLambdaWeightMultiplier weight_multiplier_;
#endif
};

class MAPLambdaWeightComputer
#if defined(__HIPCC__)
  : public IndexablePredictionSorter
#endif
{
 public:
  struct MAPStats {
    /*! \brief the accumulated precision */
    float ap_acc{0.0f};
    /*!
     * \brief the accumulated precision,
     *   assuming a positive instance is missing
     */
    float ap_acc_miss{0.0f};
    /*!
     * \brief the accumulated precision,
     * assuming that one more positive instance is inserted ahead
     */
    float ap_acc_add{0.0f};
    /* \brief the accumulated positive instance count */
    float hits{0.0f};

    XGBOOST_DEVICE MAPStats() {}  // NOLINT
    XGBOOST_DEVICE MAPStats(float ap_acc, float ap_acc_miss, float ap_acc_add, float hits)
      : ap_acc(ap_acc), ap_acc_miss(ap_acc_miss), ap_acc_add(ap_acc_add), hits(hits) {}

    // For prefix scan
    XGBOOST_DEVICE MAPStats operator +(const MAPStats &v1) const {
      return {ap_acc + v1.ap_acc, ap_acc_miss + v1.ap_acc_miss,
              ap_acc_add + v1.ap_acc_add, hits + v1.hits};
    }

    // For test purposes - compare for equality
    XGBOOST_DEVICE bool operator ==(const MAPStats &rhs) const {
      return ap_acc == rhs.ap_acc && ap_acc_miss == rhs.ap_acc_miss &&
             ap_acc_add == rhs.ap_acc_add && hits == rhs.hits;
    }
  };

 private:
  template <typename T>
  XGBOOST_DEVICE inline static void Swap(T &v0, T &v1) {
#if defined(__HIPCC__)
    thrust::swap(v0, v1);
#else
    std::swap(v0, v1);
#endif
  }

  /*!
   * \brief Obtain the delta MAP by trying to switch the positions of labels in pos_pred_pos or
   *        neg_pred_pos when sorted by predictions
   * \param pos_pred_pos positive label's prediction value position when the groups prediction
   *        values are sorted
   * \param neg_pred_pos negative label's prediction value position when the groups prediction
   *        values are sorted
   * \param pos_label, neg_label the chosen positive and negative labels
   * \param p_map_stats a vector containing the accumulated precisions for each position in a list
   * \param map_stats_size size of the accumulated precisions vector
   */
  XGBOOST_DEVICE inline static bst_float GetLambdaMAP(
    int pos_pred_pos, int neg_pred_pos,
    bst_float pos_label, bst_float neg_label,
    const MAPStats *p_map_stats, uint32_t map_stats_size) {
    if (pos_pred_pos == neg_pred_pos || p_map_stats[map_stats_size - 1].hits == 0) {
      return 0.0f;
    }
    if (pos_pred_pos > neg_pred_pos) {
      Swap(pos_pred_pos, neg_pred_pos);
      Swap(pos_label, neg_label);
    }
    bst_float original = p_map_stats[neg_pred_pos].ap_acc;
    if (pos_pred_pos != 0) original -= p_map_stats[pos_pred_pos - 1].ap_acc;
    bst_float changed = 0;
    bst_float label1 = pos_label > 0.0f ? 1.0f : 0.0f;
    bst_float label2 = neg_label > 0.0f ? 1.0f : 0.0f;
    if (label1 == label2) {
      return 0.0;
    } else if (label1 < label2) {
      changed += p_map_stats[neg_pred_pos - 1].ap_acc_add - p_map_stats[pos_pred_pos].ap_acc_add;
      changed += (p_map_stats[pos_pred_pos].hits + 1.0f) / (pos_pred_pos + 1);
    } else {
      changed += p_map_stats[neg_pred_pos - 1].ap_acc_miss - p_map_stats[pos_pred_pos].ap_acc_miss;
      changed += p_map_stats[neg_pred_pos].hits / (neg_pred_pos + 1);
    }
    bst_float ans = (changed - original) / (p_map_stats[map_stats_size - 1].hits);
    if (ans < 0) ans = -ans;
    return ans;
  }

 public:
  /*
   * \brief obtain preprocessing results for calculating delta MAP
   * \param sorted_list the list containing entry information
   * \param map_stats a vector containing the accumulated precisions for each position in a list
   */
  inline static void GetMAPStats(const std::vector<ListEntry> &sorted_list,
                                 std::vector<MAPStats> *p_map_acc) {
    std::vector<MAPStats> &map_acc = *p_map_acc;
    map_acc.resize(sorted_list.size());
    bst_float hit = 0, acc1 = 0, acc2 = 0, acc3 = 0;
    for (size_t i = 1; i <= sorted_list.size(); ++i) {
      if (sorted_list[i - 1].label > 0.0f) {
        hit++;
        acc1 += hit / i;
        acc2 += (hit - 1) / i;
        acc3 += (hit + 1) / i;
      }
      map_acc[i - 1] = MAPStats(acc1, acc2, acc3, hit);
    }
  }

  static char const* Name() {
    return "rank:map";
  }

  static void GetLambdaWeight(const std::vector<ListEntry> &sorted_list,
                              std::vector<LambdaPair> *io_pairs) {
    std::vector<LambdaPair> &pairs = *io_pairs;
    std::vector<MAPStats> map_stats;
    GetMAPStats(sorted_list, &map_stats);
    for (auto & pair : pairs) {
      pair.weight *=
        GetLambdaMAP(pair.pos_index, pair.neg_index,
                     sorted_list[pair.pos_index].label, sorted_list[pair.neg_index].label,
                     &map_stats[0], map_stats.size());
    }
  }

#if defined(__HIPCC__)
  MAPLambdaWeightComputer(const bst_float *dpreds,
                          const bst_float *dlabels,
                          const dh::SegmentSorter<float> &segment_label_sorter)
    : IndexablePredictionSorter(dpreds, segment_label_sorter),
      dmap_stats_(segment_label_sorter.GetNumItems(), MAPStats()),
      weight_multiplier_(segment_label_sorter, *this) {
    this->CreateMAPStats(dlabels, segment_label_sorter);
  }

  void CreateMAPStats(const bst_float *dlabels,
                      const dh::SegmentSorter<float> &segment_label_sorter) {
    // For each group, go through the sorted prediction positions, and look up its corresponding
    // label from the unsorted labels (from the original label list)

    // For each item in the group, compute its MAP stats.
    // Interleave the computation of map stats amongst different groups.

    // First, determine postive labels in the dataset individually
    auto nitems = segment_label_sorter.GetNumItems();
    dh::caching_device_vector<uint32_t> dhits(nitems, 0);
    // Original positions of the predictions after they have been sorted
    const auto &pred_original_pos = this->GetPredictionSorter().GetOriginalPositionsSpan();
    // Unsorted labels
    const float *unsorted_labels = dlabels;
    auto DeterminePositiveLabelLambda = [=] __device__(uint32_t idx) {
      return (unsorted_labels[pred_original_pos[idx]] > 0.0f) ? 1 : 0;
    };  // NOLINT

    thrust::transform(thrust::make_counting_iterator(static_cast<uint32_t>(0)),
                      thrust::make_counting_iterator(nitems),
                      dhits.begin(),
                      DeterminePositiveLabelLambda);

    // Allocator to be used by sort for managing space overhead while performing prefix scans
    dh::XGBCachingDeviceAllocator<char> alloc;

    // Next, prefix scan the positive labels that are segmented to accumulate them.
    // This is required for computing the accumulated precisions
    const auto &group_segments = segment_label_sorter.GetGroupSegmentsSpan();
    // Data segmented into different groups...
    thrust::inclusive_scan_by_key(thrust::cuda::par(alloc),
                                  dh::tcbegin(group_segments), dh::tcend(group_segments),
                                  dhits.begin(),  // Input value
                                  dhits.begin());  // In-place scan

    // Compute accumulated precisions for each item, assuming positive and
    // negative instances are missing.
    // But first, compute individual item precisions
    const auto *dhits_arr = dhits.data().get();
    // Group info on device
    const auto &dgroups = segment_label_sorter.GetGroupsSpan();
    auto ComputeItemPrecisionLambda = [=] __device__(uint32_t idx) {
      if (unsorted_labels[pred_original_pos[idx]] > 0.0f) {
        auto idx_within_group = (idx - dgroups[group_segments[idx]]) + 1;
        return MAPStats{static_cast<float>(dhits_arr[idx]) / idx_within_group,
                        static_cast<float>(dhits_arr[idx] - 1) / idx_within_group,
                        static_cast<float>(dhits_arr[idx] + 1) / idx_within_group,
                        1.0f};
      }
      return MAPStats{};
    };  // NOLINT

    thrust::transform(thrust::make_counting_iterator(static_cast<uint32_t>(0)),
                      thrust::make_counting_iterator(nitems),
                      this->dmap_stats_.begin(),
                      ComputeItemPrecisionLambda);

    // Lastly, compute the accumulated precisions for all the items segmented by groups.
    // The precisions are accumulated within each group
    thrust::inclusive_scan_by_key(thrust::cuda::par(alloc),
                                  dh::tcbegin(group_segments), dh::tcend(group_segments),
                                  this->dmap_stats_.begin(),  // Input map stats
                                  this->dmap_stats_.begin());  // In-place scan and output here
  }

  inline const common::Span<const MAPStats> GetMapStatsSpan() const {
    return { dmap_stats_.data().get(), dmap_stats_.size() };
  }

  // Type containing device pointers that can be cheaply copied on the kernel
  class MAPLambdaWeightMultiplier : public BaseLambdaWeightMultiplier {
   public:
    MAPLambdaWeightMultiplier(const dh::SegmentSorter<float> &segment_label_sorter,
                              const MAPLambdaWeightComputer &lwc)
      : BaseLambdaWeightMultiplier(segment_label_sorter, lwc.GetPredictionSorter()),
        dmap_stats_(lwc.GetMapStatsSpan()) {}

    // Adjust the items weight by this value
    __device__ __forceinline__ bst_float GetWeight(uint32_t gidx, int pidx, int nidx) const {
      uint32_t group_begin = dgroups_[gidx];
      uint32_t group_end = dgroups_[gidx + 1];

      auto pos_lab_orig_posn = dorig_pos_[pidx];
      auto neg_lab_orig_posn = dorig_pos_[nidx];
      KERNEL_CHECK(pos_lab_orig_posn != neg_lab_orig_posn);

      // Note: the label positive and negative indices are relative to the entire dataset.
      // Hence, scale them back to an index within the group
      auto pos_pred_pos = dindexable_sorted_preds_pos_[pos_lab_orig_posn] - group_begin;
      auto neg_pred_pos = dindexable_sorted_preds_pos_[neg_lab_orig_posn] - group_begin;
      return MAPLambdaWeightComputer::GetLambdaMAP(
        pos_pred_pos, neg_pred_pos,
        dsorted_labels_[pidx], dsorted_labels_[nidx],
        &dmap_stats_[group_begin], group_end - group_begin);
    }

   private:
    common::Span<const MAPStats> dmap_stats_;  // Start address of the map stats for every sorted
                                               // prediction value
  };

  inline const MAPLambdaWeightMultiplier GetWeightMultiplier() const { return weight_multiplier_; }

 private:
  dh::caching_device_vector<MAPStats> dmap_stats_;
  // This computes the adjustment to the weight
  const MAPLambdaWeightMultiplier weight_multiplier_;
#endif
};

#if defined(__HIPCC__)
class SortedLabelList : dh::SegmentSorter<float> {
 private:
  const LambdaRankParam &param_;                      // Objective configuration

 public:
  explicit SortedLabelList(const LambdaRankParam &param)
    : param_(param) {}

  // Sort the labels that are grouped by 'groups'
  void Sort(const HostDeviceVector<bst_float> &dlabels, const std::vector<uint32_t> &groups) {
    this->SortItems(dlabels.ConstDevicePointer(), dlabels.Size(), groups);
  }

  // This kernel can only run *after* the kernel in sort is completed, as they
  // use the default stream
  template <typename LambdaWeightComputerT>
  void ComputeGradients(const bst_float *dpreds,   // Unsorted predictions
                        const bst_float *dlabels,  // Unsorted labels
                        const HostDeviceVector<bst_float> &weights,
                        int iter,
                        GradientPair *out_gpair,
                        float weight_normalization_factor) {
    // Group info on device
    const auto &dgroups = this->GetGroupsSpan();
    uint32_t ngroups = this->GetNumGroups() + 1;

    uint32_t total_items = this->GetNumItems();
    uint32_t niter = param_.num_pairsample * total_items;

    float fix_list_weight = param_.fix_list_weight;

    const auto &original_pos = this->GetOriginalPositionsSpan();

    uint32_t num_weights = weights.Size();
    auto dweights = num_weights ? weights.ConstDevicePointer() : nullptr;

    const auto &sorted_labels = this->GetItemsSpan();

    // This is used to adjust the weight of different elements based on the different ranking
    // objective function policies
    LambdaWeightComputerT weight_computer(dpreds, dlabels, *this);
    auto wmultiplier = weight_computer.GetWeightMultiplier();

    int device_id = -1;
    dh::safe_cuda(hipGetDevice(&device_id));
    // For each instance in the group, compute the gradient pair concurrently
    dh::LaunchN(device_id, niter, nullptr, [=] __device__(uint32_t idx) {
      // First, determine the group 'idx' belongs to
      uint32_t item_idx = idx % total_items;
      uint32_t group_idx =
          thrust::upper_bound(thrust::seq, dgroups.begin(),
                              dgroups.begin() + ngroups, item_idx) -
          dgroups.begin();
      // Span of this group within the larger labels/predictions sorted tuple
      uint32_t group_begin = dgroups[group_idx - 1];
      uint32_t group_end = dgroups[group_idx];
      uint32_t total_group_items = group_end - group_begin;

      // Are the labels diverse enough? If they are all the same, then there is nothing to pick
      // from another group - bail sooner
      if (sorted_labels[group_begin] == sorted_labels[group_end - 1]) return;

      // Find the number of labels less than and greater than the current label
      // at the sorted index position item_idx
      uint32_t nleft  = CountNumItemsToTheLeftOf(
        sorted_labels.data() + group_begin, item_idx - group_begin + 1, sorted_labels[item_idx]);
      uint32_t nright = CountNumItemsToTheRightOf(
        sorted_labels.data() + item_idx, group_end - item_idx, sorted_labels[item_idx]);

      // Create a minstd_rand object to act as our source of randomness
      thrust::minstd_rand rng((iter + 1) * 1111);
      rng.discard(((idx / total_items) * total_group_items) + item_idx - group_begin);
      // Create a uniform_int_distribution to produce a sample from outside of the
      // present label group
      thrust::uniform_int_distribution<int> dist(0, nleft + nright - 1);

      int sample = dist(rng);
      int pos_idx = -1;  // Bigger label
      int neg_idx = -1;  // Smaller label
      // Are we picking a sample to the left/right of the current group?
      if (sample < nleft) {
        // Go left
        pos_idx = sample + group_begin;
        neg_idx = item_idx;
      } else {
        pos_idx = item_idx;
        uint32_t items_in_group = total_group_items - nleft - nright;
        neg_idx = sample + items_in_group + group_begin;
      }

      // Compute and assign the gradients now
      const float eps = 1e-16f;
      bst_float p = common::Sigmoid(dpreds[original_pos[pos_idx]] - dpreds[original_pos[neg_idx]]);
      bst_float g = p - 1.0f;
      bst_float h = thrust::max(p * (1.0f - p), eps);

      // Rescale each gradient and hessian so that the group has a weighted constant
      float scale = __frcp_ru(niter / total_items);
      if (fix_list_weight != 0.0f) {
        scale *= fix_list_weight / total_group_items;
      }

      float weight = num_weights ? dweights[group_idx - 1] : 1.0f;
      weight *= weight_normalization_factor;
      weight *= wmultiplier.GetWeight(group_idx - 1, pos_idx, neg_idx);
      weight *= scale;
      // Accumulate gradient and hessian in both positive and negative indices
      const GradientPair in_pos_gpair(g * weight, 2.0f * weight * h);
      dh::AtomicAddGpair(&out_gpair[original_pos[pos_idx]], in_pos_gpair);

      const GradientPair in_neg_gpair(-g * weight, 2.0f * weight * h);
      dh::AtomicAddGpair(&out_gpair[original_pos[neg_idx]], in_neg_gpair);
    });

    // Wait until the computations done by the kernel is complete
    dh::safe_cuda(hipStreamSynchronize(nullptr));
  }
};
#endif

// objective for lambda rank
template <typename LambdaWeightComputerT>
class LambdaRankObj : public ObjFunction {
 public:
  void Configure(const std::vector<std::pair<std::string, std::string> >& args) override {
    param_.UpdateAllowUnknown(args);
  }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo& info,
                   int iter,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    CHECK_EQ(preds.Size(), info.labels_.Size()) << "label size predict size not match";

    // quick consistency when group is not available
    std::vector<unsigned> tgptr(2, 0); tgptr[1] = static_cast<unsigned>(info.labels_.Size());
    const std::vector<unsigned> &gptr = info.group_ptr_.size() == 0 ? tgptr : info.group_ptr_;
    CHECK(gptr.size() != 0 && gptr.back() == info.labels_.Size())
          << "group structure not consistent with #rows" << ", "
          << "group ponter size: " << gptr.size() << ", "
          << "labels size: " << info.labels_.Size() << ", "
          << "group pointer back: " << (gptr.size() == 0 ? 0 : gptr.back());

#if defined(__HIPCC__)
    // Check if we have a GPU assignment; else, revert back to CPU
    auto device = tparam_->gpu_id;
    if (device >= 0) {
      ComputeGradientsOnGPU(preds, info, iter, out_gpair, gptr);
    } else {
      // Revert back to CPU
#endif
      ComputeGradientsOnCPU(preds, info, iter, out_gpair, gptr);
#if defined(__HIPCC__)
    }
#endif
  }

  const char* DefaultEvalMetric() const override {
    return "map";
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String(LambdaWeightComputerT::Name());
    out["lambda_rank_param"] = ToJson(param_);
  }

  void LoadConfig(Json const& in) override {
    FromJson(in["lambda_rank_param"], &param_);
  }

 private:
  bst_float ComputeWeightNormalizationFactor(const MetaInfo& info,
                                             const std::vector<unsigned> &gptr) {
    const auto ngroup = static_cast<bst_omp_uint>(gptr.size() - 1);
    bst_float sum_weights = 0;
    for (bst_omp_uint k = 0; k < ngroup; ++k) {
      sum_weights += info.GetWeight(k);
    }
    return ngroup / sum_weights;
  }

  void ComputeGradientsOnCPU(const HostDeviceVector<bst_float>& preds,
                             const MetaInfo& info,
                             int iter,
                             HostDeviceVector<GradientPair>* out_gpair,
                             const std::vector<unsigned> &gptr) {
    LOG(DEBUG) << "Computing " << LambdaWeightComputerT::Name() << " gradients on CPU.";

    bst_float weight_normalization_factor = ComputeWeightNormalizationFactor(info, gptr);

    const auto& preds_h = preds.HostVector();
    const auto& labels = info.labels_.HostVector();
    std::vector<GradientPair>& gpair = out_gpair->HostVector();
    const auto ngroup = static_cast<bst_omp_uint>(gptr.size() - 1);
    out_gpair->Resize(preds.Size());

    #pragma omp parallel
    {
      // parallel construct, declare random number generator here, so that each
      // thread use its own random number generator, seed by thread id and current iteration
      std::minstd_rand rnd((iter + 1) * 1111);
      std::vector<LambdaPair> pairs;
      std::vector<ListEntry>  lst;
      std::vector< std::pair<bst_float, unsigned> > rec;

      #pragma omp for schedule(static)
      for (bst_omp_uint k = 0; k < ngroup; ++k) {
        lst.clear(); pairs.clear();
        for (unsigned j = gptr[k]; j < gptr[k+1]; ++j) {
          lst.emplace_back(preds_h[j], labels[j], j);
          gpair[j] = GradientPair(0.0f, 0.0f);
        }
        std::stable_sort(lst.begin(), lst.end(), ListEntry::CmpPred);
        rec.resize(lst.size());
        for (unsigned i = 0; i < lst.size(); ++i) {
          rec[i] = std::make_pair(lst[i].label, i);
        }
        std::stable_sort(rec.begin(), rec.end(), common::CmpFirst);
        // enumerate buckets with same label, for each item in the lst, grab another sample randomly
        for (unsigned i = 0; i < rec.size(); ) {
          unsigned j = i + 1;
          while (j < rec.size() && rec[j].first == rec[i].first) ++j;
          // bucket in [i,j), get a sample outside bucket
          unsigned nleft = i, nright = static_cast<unsigned>(rec.size() - j);
          if (nleft + nright != 0) {
            int nsample = param_.num_pairsample;
            while (nsample --) {
              for (unsigned pid = i; pid < j; ++pid) {
                unsigned ridx = std::uniform_int_distribution<unsigned>(0, nleft + nright - 1)(rnd);
                if (ridx < nleft) {
                  pairs.emplace_back(rec[ridx].second, rec[pid].second,
                      info.GetWeight(k) * weight_normalization_factor);
                } else {
                  pairs.emplace_back(rec[pid].second, rec[ridx+j-i].second,
                      info.GetWeight(k) * weight_normalization_factor);
                }
              }
            }
          }
          i = j;
        }
        // get lambda weight for the pairs
        LambdaWeightComputerT::GetLambdaWeight(lst, &pairs);
        // rescale each gradient and hessian so that the lst have constant weighted
        float scale = 1.0f / param_.num_pairsample;
        if (param_.fix_list_weight != 0.0f) {
          scale *= param_.fix_list_weight / (gptr[k + 1] - gptr[k]);
        }
        for (auto & pair : pairs) {
          const ListEntry &pos = lst[pair.pos_index];
          const ListEntry &neg = lst[pair.neg_index];
          const bst_float w = pair.weight * scale;
          const float eps = 1e-16f;
          bst_float p = common::Sigmoid(pos.pred - neg.pred);
          bst_float g = p - 1.0f;
          bst_float h = std::max(p * (1.0f - p), eps);
          // accumulate gradient and hessian in both pid, and nid
          gpair[pos.rindex] += GradientPair(g * w, 2.0f*w*h);
          gpair[neg.rindex] += GradientPair(-g * w, 2.0f*w*h);
        }
      }
    }
  }

#if defined(__HIPCC__)
  void ComputeGradientsOnGPU(const HostDeviceVector<bst_float>& preds,
                             const MetaInfo& info,
                             int iter,
                             HostDeviceVector<GradientPair>* out_gpair,
                             const std::vector<unsigned> &gptr) {
    LOG(DEBUG) << "Computing " << LambdaWeightComputerT::Name() << " gradients on GPU.";

    auto device = tparam_->gpu_id;
    dh::safe_cuda(hipSetDevice(device));

    bst_float weight_normalization_factor = ComputeWeightNormalizationFactor(info, gptr);

    // Set the device ID and copy them to the device
    out_gpair->SetDevice(device);
    info.labels_.SetDevice(device);
    preds.SetDevice(device);
    info.weights_.SetDevice(device);

    out_gpair->Resize(preds.Size());

    auto d_preds = preds.ConstDevicePointer();
    auto d_gpair = out_gpair->DevicePointer();
    auto d_labels = info.labels_.ConstDevicePointer();

    SortedLabelList slist(param_);

    // Sort the labels within the groups on the device
    slist.Sort(info.labels_, gptr);

    // Initialize the gradients next
    out_gpair->Fill(GradientPair(0.0f, 0.0f));

    // Finally, compute the gradients
    slist.ComputeGradients<LambdaWeightComputerT>
      (d_preds, d_labels, info.weights_, iter, d_gpair, weight_normalization_factor);
  }
#endif

  LambdaRankParam param_;
};

#if !defined(GTEST_TEST)
// register the objective functions
DMLC_REGISTER_PARAMETER(LambdaRankParam);

XGBOOST_REGISTER_OBJECTIVE(PairwiseRankObj, PairwiseLambdaWeightComputer::Name())
.describe("Pairwise rank objective.")
.set_body([]() { return new LambdaRankObj<PairwiseLambdaWeightComputer>(); });

XGBOOST_REGISTER_OBJECTIVE(LambdaRankNDCG, NDCGLambdaWeightComputer::Name())
.describe("LambdaRank with NDCG as objective.")
.set_body([]() { return new LambdaRankObj<NDCGLambdaWeightComputer>(); });

XGBOOST_REGISTER_OBJECTIVE(LambdaRankObjMAP, MAPLambdaWeightComputer::Name())
.describe("LambdaRank with MAP as objective.")
.set_body([]() { return new LambdaRankObj<MAPLambdaWeightComputer>(); });
#endif

}  // namespace obj
}  // namespace xgboost
