#include "hip/hip_runtime.h"
/*!
 * Copyright 2017 XGBoost contributors
 */
// GPU implementation of objective function.
// Necessary to avoid extra copying of data to CPU.
#include <dmlc/omp.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <xgboost/logging.h>
#include <xgboost/objective.h>
#include <cmath>
#include <memory>
#include <vector>

#include "../common/device_helpers.cuh"
#include "../common/host_device_vector.h"
#include "./regression_loss.h"


namespace xgboost {
namespace obj {

using dh::DVec;

DMLC_REGISTRY_FILE_TAG(regression_obj_gpu);

struct GPURegLossParam : public dmlc::Parameter<GPURegLossParam> {
  float scale_pos_weight;
  int n_gpus;
  int gpu_id;
  // declare parameters
  DMLC_DECLARE_PARAMETER(GPURegLossParam) {
    DMLC_DECLARE_FIELD(scale_pos_weight).set_default(1.0f).set_lower_bound(0.0f)
      .describe("Scale the weight of positive examples by this factor");
    DMLC_DECLARE_FIELD(n_gpus).set_default(1).set_lower_bound(-1)
      .describe("Number of GPUs to use for multi-gpu algorithms (NOT IMPLEMENTED)");
    DMLC_DECLARE_FIELD(gpu_id)
      .set_lower_bound(0)
      .set_default(0)
      .describe("gpu to use for objective function evaluation");
  }
};

// GPU kernel for gradient computation
template<typename Loss>
__global__ void get_gradient_k
(GradientPair *__restrict__ out_gpair,  unsigned int *__restrict__ label_correct,
 const float * __restrict__ preds, const float * __restrict__ labels,
 const float * __restrict__ weights, int n, float scale_pos_weight) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= n)
    return;
  float p = Loss::PredTransform(preds[i]);
  float w = weights == nullptr ? 1.0f : weights[i];
  float label = labels[i];
  if (label == 1.0f)
    w *= scale_pos_weight;
  if (!Loss::CheckLabel(label))
    atomicAnd(label_correct, 0);
  out_gpair[i] = GradientPair
    (Loss::FirstOrderGradient(p, label) * w, Loss::SecondOrderGradient(p, label) * w);
}

// GPU kernel for predicate transformation
template<typename Loss>
__global__ void pred_transform_k(float * __restrict__ preds, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= n)
    return;
  preds[i] = Loss::PredTransform(preds[i]);
}

// regression loss function for evaluation on GPU (eventually)
template<typename Loss>
class GPURegLossObj : public ObjFunction {
 protected:
  bool copied_;
  HostDeviceVector<bst_float> labels_, weights_;
  HostDeviceVector<unsigned int> label_correct_;

  // allocate device data for n elements, do nothing if memory is allocated already
  void LazyResize(size_t n, size_t n_weights) {
    if (labels_.Size() == n && weights_.Size() == n_weights)
      return;
    copied_ = false;

    labels_.Reshard(devices_);
    weights_.Reshard(devices_);
    label_correct_.Reshard(devices_);

    if (labels_.Size() != n) {
      labels_.Resize(n);
      label_correct_.Resize(devices_.Size());
    }
    if (weights_.Size() != n_weights)
      weights_.Resize(n_weights);
  }

 public:
  GPURegLossObj() : copied_(false) {}

  void Configure(const std::vector<std::pair<std::string, std::string> >& args) override {
    param_.InitAllowUnknown(args);
    CHECK(param_.n_gpus != 0) << "Must have at least one device";
    devices_ = GPUSet::Range(param_.gpu_id, dh::NDevicesAll(param_.n_gpus));
  }

  void GetGradient(HostDeviceVector<float>* preds,
                   const MetaInfo &info,
                   int iter,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    CHECK_NE(info.labels_.size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds->Size(), info.labels_.size())
      << "labels are not correctly provided"
      << "preds.size=" << preds->Size() << ", label.size=" << info.labels_.size();
    size_t ndata = preds->Size();
    preds->Reshard(devices_);
    out_gpair->Reshard(devices_);
    out_gpair->Resize(ndata);
    LazyResize(ndata, info.weights_.size());
    GetGradientDevice(preds, info, iter, out_gpair);
  }

 private:
  void GetGradientDevice(HostDeviceVector<float>* preds,
                         const MetaInfo &info,
                         int iter,
                         HostDeviceVector<GradientPair>* out_gpair) {
    label_correct_.Fill(1);
    // only copy the labels and weights once, similar to how the data is copied
    if (!copied_) {
      labels_.Copy(info.labels_);
      if (info.weights_.size() > 0)
        weights_.Copy(info.weights_);
      copied_ = true;
    }

    // run the kernel
#pragma omp parallel for schedule(static, 1) if (devices_.Size() > 1)
    for (int i = 0; i < devices_.Size(); ++i) {
      int d = devices_[i];
      dh::safe_cuda(hipSetDevice(d));
      const int block = 256;
      size_t n = preds->DeviceSize(d);
      if (n > 0) {
        get_gradient_k<Loss><<<dh::DivRoundUp(n, block), block>>>
          (out_gpair->DevicePointer(d), label_correct_.DevicePointer(d),
           preds->DevicePointer(d), labels_.DevicePointer(d),
           info.weights_.size() > 0 ? weights_.DevicePointer(d) : nullptr,
           n, param_.scale_pos_weight);
        dh::safe_cuda(hipGetLastError());
      }
      dh::safe_cuda(hipDeviceSynchronize());
    }

    // copy "label correct" flags back to host
    std::vector<unsigned int>& label_correct_h = label_correct_.HostVector();
    for (int i = 0; i < devices_.Size(); ++i) {
      if (label_correct_h[i] == 0)
        LOG(FATAL) << Loss::LabelErrorMsg();
    }
  }

 public:
  const char* DefaultEvalMetric() const override {
    return Loss::DefaultEvalMetric();
  }

  void PredTransform(HostDeviceVector<float> *io_preds) override {
    io_preds->Reshard(devices_);
    size_t ndata = io_preds->Size();
    PredTransformDevice(io_preds);
  }

  void PredTransformDevice(HostDeviceVector<float>* preds) {
#pragma omp parallel for schedule(static, 1) if (devices_.Size() > 1)
    for (int i = 0; i < devices_.Size(); ++i) {
      int d = devices_[i];
      dh::safe_cuda(hipSetDevice(d));
      const int block = 256;
      size_t n = preds->DeviceSize(d);
      if (n > 0) {
        pred_transform_k<Loss><<<dh::DivRoundUp(n, block), block>>>(preds->DevicePointer(d), n);
        dh::safe_cuda(hipGetLastError());
      }
      dh::safe_cuda(hipDeviceSynchronize());
    }
  }

  float ProbToMargin(float base_score) const override {
    return Loss::ProbToMargin(base_score);
  }

 protected:
  GPURegLossParam param_;
  GPUSet devices_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(GPURegLossParam);

XGBOOST_REGISTER_OBJECTIVE(GPULinearRegression, "gpu:reg:linear")
.describe("Linear regression (computed on GPU).")
.set_body([]() { return new GPURegLossObj<LinearSquareLoss>(); });

XGBOOST_REGISTER_OBJECTIVE(GPULogisticRegression, "gpu:reg:logistic")
.describe("Logistic regression for probability regression task (computed on GPU).")
.set_body([]() { return new GPURegLossObj<LogisticRegression>(); });

XGBOOST_REGISTER_OBJECTIVE(GPULogisticClassification, "gpu:binary:logistic")
.describe("Logistic regression for binary classification task (computed on GPU).")
.set_body([]() { return new GPURegLossObj<LogisticClassification>(); });

XGBOOST_REGISTER_OBJECTIVE(GPULogisticRaw, "gpu:binary:logitraw")
.describe("Logistic regression for classification, output score "
          "before logistic transformation (computed on GPU)")
.set_body([]() { return new GPURegLossObj<LogisticRaw>(); });

}  // namespace obj
}  // namespace xgboost
