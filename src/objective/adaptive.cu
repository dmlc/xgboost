/**
 * Copyright 2022-2023 by XGBoost Contributors
 */
#include <thrust/sort.h>

#include <cstdint>                     // std::int32_t
#include <hipcub/hipcub.hpp>                 // NOLINT

#include "../common/cuda_context.cuh"  // HIPContext
#include "../common/device_helpers.cuh"
#include "../common/stats.cuh"
#include "adaptive.h"
#include "xgboost/context.h"

namespace xgboost {
namespace obj {
namespace detail {
void EncodeTreeLeafDevice(Context const* ctx, common::Span<bst_node_t const> position,
                          dh::device_vector<size_t>* p_ridx, HostDeviceVector<size_t>* p_nptr,
                          HostDeviceVector<bst_node_t>* p_nidx, RegTree const& tree) {
  // copy position to buffer
  dh::safe_cuda(hipSetDevice(ctx->gpu_id));
  auto cuctx = ctx->CUDACtx();
  size_t n_samples = position.size();
  dh::device_vector<bst_node_t> sorted_position(position.size());
  dh::safe_cuda(hipMemcpyAsync(sorted_position.data().get(), position.data(),
                                position.size_bytes(), hipMemcpyDeviceToDevice, cuctx->Stream()));

  p_ridx->resize(position.size());
  dh::Iota(dh::ToSpan(*p_ridx));
  // sort row index according to node index
  thrust::stable_sort_by_key(cuctx->TP(), sorted_position.begin(),
                             sorted_position.begin() + n_samples, p_ridx->begin());
  size_t beg_pos =
      thrust::find_if(cuctx->CTP(), sorted_position.cbegin(), sorted_position.cend(),
                      [] XGBOOST_DEVICE(bst_node_t nidx) { return nidx >= 0; }) -
      sorted_position.cbegin();
  if (beg_pos == sorted_position.size()) {
    auto& leaf = p_nidx->HostVector();
    tree.WalkTree([&](bst_node_t nidx) {
      if (tree[nidx].IsLeaf()) {
        leaf.push_back(nidx);
      }
      return true;
    });
    return;
  }

  size_t n_leaf = tree.GetNumLeaves();
  size_t max_n_unique = n_leaf;

  dh::caching_device_vector<size_t> counts_out(max_n_unique + 1, 0);
  auto d_counts_out = dh::ToSpan(counts_out).subspan(0, max_n_unique);
  auto d_num_runs_out = dh::ToSpan(counts_out).subspan(max_n_unique, 1);
  dh::caching_device_vector<bst_node_t> unique_out(max_n_unique, 0);
  auto d_unique_out = dh::ToSpan(unique_out);

  size_t nbytes{0};
  auto begin_it = sorted_position.begin() + beg_pos;
  dh::safe_cuda(hipcub::DeviceRunLengthEncode::Encode(
      nullptr, nbytes, begin_it, unique_out.data().get(), counts_out.data().get(),
      d_num_runs_out.data(), n_samples - beg_pos, ctx->CUDACtx()->Stream()));
  dh::TemporaryArray<char> temp(nbytes);
  dh::safe_cuda(hipcub::DeviceRunLengthEncode::Encode(
      temp.data().get(), nbytes, begin_it, unique_out.data().get(), counts_out.data().get(),
      d_num_runs_out.data(), n_samples - beg_pos, ctx->CUDACtx()->Stream()));

  dh::PinnedMemory pinned_pool;
  auto pinned = pinned_pool.GetSpan<char>(sizeof(size_t) + sizeof(bst_node_t));
  dh::CUDAStream copy_stream;
  size_t* h_num_runs = reinterpret_cast<size_t*>(pinned.subspan(0, sizeof(size_t)).data());

  dh::CUDAEvent e;
  e.Record(cuctx->Stream());
  copy_stream.View().Wait(e);
  // flag for whether there's ignored position
  bst_node_t* h_first_unique =
      reinterpret_cast<bst_node_t*>(pinned.subspan(sizeof(size_t), sizeof(bst_node_t)).data());
  dh::safe_cuda(hipMemcpyAsync(h_num_runs, d_num_runs_out.data(), sizeof(size_t),
                                hipMemcpyDeviceToHost, copy_stream.View()));
  dh::safe_cuda(hipMemcpyAsync(h_first_unique, d_unique_out.data(), sizeof(bst_node_t),
                                hipMemcpyDeviceToHost, copy_stream.View()));

  /**
   * copy node index (leaf index)
   */
  auto& nidx = *p_nidx;
  auto& nptr = *p_nptr;
  nidx.SetDevice(ctx->gpu_id);
  nidx.Resize(n_leaf);
  auto d_node_idx = nidx.DeviceSpan();

  nptr.SetDevice(ctx->gpu_id);
  nptr.Resize(n_leaf + 1, 0);
  auto d_node_ptr = nptr.DeviceSpan();

  dh::LaunchN(n_leaf, [=] XGBOOST_DEVICE(size_t i) {
    if (i >= d_num_runs_out[0]) {
      // d_num_runs_out <= max_n_unique
      // this omits all the leaf that are empty. A leaf can be empty when there's
      // missing data, which can be caused by sparse input and distributed training.
      return;
    }
    d_node_idx[i] = d_unique_out[i];
    d_node_ptr[i + 1] = d_counts_out[i];
    if (i == 0) {
      d_node_ptr[0] = beg_pos;
    }
  });
  thrust::inclusive_scan(cuctx->CTP(), dh::tbegin(d_node_ptr), dh::tend(d_node_ptr),
                         dh::tbegin(d_node_ptr));
  copy_stream.View().Sync();
  CHECK_GT(*h_num_runs, 0);
  CHECK_LE(*h_num_runs, n_leaf);

  if (*h_num_runs < n_leaf) {
    // shrink to omit the sampled nodes.
    nptr.Resize(*h_num_runs + 1);
    nidx.Resize(*h_num_runs);

    std::vector<bst_node_t> leaves;
    tree.WalkTree([&](bst_node_t nidx) {
      if (tree[nidx].IsLeaf()) {
        leaves.push_back(nidx);
      }
      return true;
    });
    CHECK_EQ(leaves.size(), n_leaf);
    // Fill all the leaves that don't have any sample. This is hacky and inefficient. An
    // alternative is to leave the objective to handle missing leaf, which is more messy
    // as we need to take other distributed workers into account.
    auto& h_nidx = nidx.HostVector();
    auto& h_nptr = nptr.HostVector();
    FillMissingLeaf(leaves, &h_nidx, &h_nptr);
    nidx.DevicePointer();
    nptr.DevicePointer();
  }
  CHECK_EQ(nidx.Size(), n_leaf);
  CHECK_EQ(nptr.Size(), n_leaf + 1);
}

void UpdateTreeLeafDevice(Context const* ctx, common::Span<bst_node_t const> position,
                          std::int32_t group_idx, MetaInfo const& info, float learning_rate,
                          HostDeviceVector<float> const& predt, float alpha, RegTree* p_tree) {
  dh::safe_cuda(hipSetDevice(ctx->gpu_id));
  dh::device_vector<size_t> ridx;
  HostDeviceVector<size_t> nptr;
  HostDeviceVector<bst_node_t> nidx;

  EncodeTreeLeafDevice(ctx, position, &ridx, &nptr, &nidx, *p_tree);

  if (nptr.Empty()) {
    std::vector<float> quantiles;
    UpdateLeafValues(&quantiles, nidx.ConstHostVector(), info, learning_rate, p_tree);
  }

  HostDeviceVector<float> quantiles;
  predt.SetDevice(ctx->gpu_id);

  auto d_predt = linalg::MakeTensorView(ctx, predt.ConstDeviceSpan(), info.num_row_,
                                        predt.Size() / info.num_row_);
  CHECK_LT(group_idx, d_predt.Shape(1));
  auto t_predt = d_predt.Slice(linalg::All(), group_idx);
  auto d_labels = info.labels.View(ctx->gpu_id).Slice(linalg::All(), IdxY(info, group_idx));

  auto d_row_index = dh::ToSpan(ridx);
  auto seg_beg = nptr.DevicePointer();
  auto seg_end = seg_beg + nptr.Size();
  auto val_beg = dh::MakeTransformIterator<float>(thrust::make_counting_iterator(0ul),
                                                  [=] XGBOOST_DEVICE(size_t i) {
                                                    float p = t_predt(d_row_index[i]);
                                                    auto y = d_labels(d_row_index[i]);
                                                    return y - p;
                                                  });
  CHECK_EQ(d_labels.Shape(0), position.size());
  auto val_end = val_beg + d_labels.Shape(0);
  CHECK_EQ(nidx.Size() + 1, nptr.Size());
  if (info.weights_.Empty()) {
    common::SegmentedQuantile(ctx, alpha, seg_beg, seg_end, val_beg, val_end, &quantiles);
  } else {
    info.weights_.SetDevice(ctx->gpu_id);
    auto d_weights = info.weights_.ConstDeviceSpan();
    CHECK_EQ(d_weights.size(), d_row_index.size());
    auto w_it = thrust::make_permutation_iterator(dh::tcbegin(d_weights), dh::tcbegin(d_row_index));
    common::SegmentedWeightedQuantile(ctx, alpha, seg_beg, seg_end, val_beg, val_end, w_it,
                                      w_it + d_weights.size(), &quantiles);
  }

  UpdateLeafValues(&quantiles.HostVector(), nidx.ConstHostVector(), info, learning_rate, p_tree);
}
}  // namespace detail
}  // namespace obj
}  // namespace xgboost
