#include "hip/hip_runtime.h"
/*!
 * Copyright 2015-2022 by XGBoost Contributors
 * \file multi_class.cc
 * \brief Definition of multi-class classification objectives.
 * \author Tianqi Chen
 */
#include <dmlc/omp.h>

#include <vector>
#include <algorithm>
#include <limits>
#include <utility>

#include "xgboost/parameter.h"
#include "xgboost/data.h"
#include "xgboost/logging.h"
#include "xgboost/objective.h"
#include "xgboost/json.h"

#include "../common/common.h"
#include "../common/math.h"
#include "../common/transform.h"

namespace xgboost {
namespace obj {

#if defined(XGBOOST_USE_CUDA)
DMLC_REGISTRY_FILE_TAG(multiclass_obj_gpu);
#endif  // defined(XGBOOST_USE_CUDA)

struct SoftmaxMultiClassParam : public XGBoostParameter<SoftmaxMultiClassParam> {
  int num_class;
  // declare parameters
  DMLC_DECLARE_PARAMETER(SoftmaxMultiClassParam) {
    DMLC_DECLARE_FIELD(num_class).set_lower_bound(1)
        .describe("Number of output class in the multi-class classification.");
  }
};

class SoftmaxMultiClassObj : public ObjFunction {
 public:
  explicit SoftmaxMultiClassObj(bool output_prob)
  : output_prob_(output_prob) {}

  void Configure(Args const& args) override {
    param_.UpdateAllowUnknown(args);
  }

  ObjInfo Task() const override { return ObjInfo::kClassification; }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo& info,
                   int iter,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    // Remove unused parameter compiler warning.
    (void) iter;

    if (info.labels.Size() == 0) {
      return;
    }
    CHECK(preds.Size() == (static_cast<size_t>(param_.num_class) * info.labels.Size()))
        << "SoftmaxMultiClassObj: label size and pred size does not match.\n"
        << "label.Size() * num_class: "
        << info.labels.Size() * static_cast<size_t>(param_.num_class) << "\n"
        << "num_class: " << param_.num_class << "\n"
        << "preds.Size(): " << preds.Size();

    const int nclass = param_.num_class;
    const auto ndata = static_cast<int64_t>(preds.Size() / nclass);

    auto device = ctx_->gpu_id;
    out_gpair->SetDevice(device);
    info.labels.SetDevice(device);
    info.weights_.SetDevice(device);
    preds.SetDevice(device);

    label_correct_.Resize(1);
    label_correct_.SetDevice(device);

    out_gpair->Resize(preds.Size());
    label_correct_.Fill(1);

    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }

    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t idx,
                           common::Span<GradientPair> gpair,
                           common::Span<bst_float const> labels,
                           common::Span<bst_float const> preds,
                           common::Span<bst_float const> weights,
                           common::Span<int> _label_correct) {
          common::Span<bst_float const> point = preds.subspan(idx * nclass, nclass);

          // Part of Softmax function
          bst_float wmax = std::numeric_limits<bst_float>::min();
          for (auto const i : point) { wmax = fmaxf(i, wmax); }
          double wsum = 0.0f;
          for (auto const i : point) { wsum += expf(i - wmax); }
          auto label = labels[idx];
          if (label < 0 || label >= nclass) {
            _label_correct[0] = 0;
            label = 0;
          }
          bst_float wt = is_null_weight ? 1.0f : weights[idx];
          for (int k = 0; k < nclass; ++k) {
            // Computation duplicated to avoid creating a cache.
            bst_float p = expf(point[k] - wmax) / static_cast<float>(wsum);
            const float eps = 1e-16f;
            const bst_float h = fmax(2.0f * p * (1.0f - p) * wt, eps);
            p = label == k ? p - 1.0f : p;
            gpair[idx * nclass + k] = GradientPair(p * wt, h);
          }
        }, common::Range{0, ndata}, ctx_->Threads(), device)
        .Eval(out_gpair, info.labels.Data(), &preds, &info.weights_, &label_correct_);

    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag != 1) {
        LOG(FATAL) << "SoftmaxMultiClassObj: label must be in [0, num_class).";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float>* io_preds) const override {
    this->Transform(io_preds, output_prob_);
  }
  void EvalTransform(HostDeviceVector<bst_float>* io_preds) override {
    this->Transform(io_preds, true);
  }
  const char* DefaultEvalMetric() const override {
    return "mlogloss";
  }

  inline void Transform(HostDeviceVector<bst_float> *io_preds, bool prob) const {
    const int nclass = param_.num_class;
    const auto ndata = static_cast<int64_t>(io_preds->Size() / nclass);

    auto device = io_preds->DeviceIdx();
    if (prob) {
      common::Transform<>::Init(
          [=] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
            common::Span<bst_float> point =
                _preds.subspan(_idx * nclass, nclass);
            common::Softmax(point.begin(), point.end());
          },
          common::Range{0, ndata}, this->ctx_->Threads(), device)
          .Eval(io_preds);
    } else {
      io_preds->SetDevice(device);
      HostDeviceVector<bst_float> max_preds;
      max_preds.SetDevice(device);
      max_preds.Resize(ndata);
      common::Transform<>::Init(
          [=] XGBOOST_DEVICE(size_t _idx, common::Span<const bst_float> _preds,
                             common::Span<bst_float> _max_preds) {
            common::Span<const bst_float> point =
                _preds.subspan(_idx * nclass, nclass);
            _max_preds[_idx] =
                common::FindMaxIndex(point.cbegin(), point.cend()) -
                point.cbegin();
          },
          common::Range{0, ndata}, this->ctx_->Threads(), device)
          .Eval(io_preds, &max_preds);
      io_preds->Resize(max_preds.Size());
      io_preds->Copy(max_preds);
    }
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    if (this->output_prob_) {
      out["name"] = String("multi:softprob");
    } else {
      out["name"] = String("multi:softmax");
    }
    out["softmax_multiclass_param"] = ToJson(param_);
  }

  void LoadConfig(Json const& in) override {
    FromJson(in["softmax_multiclass_param"], &param_);
  }

 private:
  // output probability
  bool output_prob_;
  // parameter
  SoftmaxMultiClassParam param_;
  // Cache for max_preds
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(SoftmaxMultiClassParam);

XGBOOST_REGISTER_OBJECTIVE(SoftmaxMultiClass, "multi:softmax")
.describe("Softmax for multi-class classification, output class index.")
.set_body([]() { return new SoftmaxMultiClassObj(false); });

XGBOOST_REGISTER_OBJECTIVE(SoftprobMultiClass, "multi:softprob")
.describe("Softmax for multi-class classification, output probability distribution.")
.set_body([]() { return new SoftmaxMultiClassObj(true); });

}  // namespace obj
}  // namespace xgboost
