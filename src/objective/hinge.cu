#include "hip/hip_runtime.h"
/*!
 * Copyright 2018-2022 by XGBoost Contributors
 * \file hinge.cc
 * \brief Provides an implementation of the hinge loss function
 * \author Henry Gouk
 */
#include "xgboost/objective.h"
#include "xgboost/json.h"
#include "xgboost/span.h"
#include "xgboost/host_device_vector.h"

#include "../common/math.h"
#include "../common/transform.h"
#include "../common/common.h"

namespace xgboost {
namespace obj {

#if defined(XGBOOST_USE_CUDA)
DMLC_REGISTRY_FILE_TAG(hinge_obj_gpu);
#endif  // defined(XGBOOST_USE_CUDA)

class HingeObj : public ObjFunction {
 public:
  HingeObj() = default;

  void Configure(Args const&) override {}
  ObjInfo Task() const override { return ObjInfo::kRegression; }

  void GetGradient(const HostDeviceVector<bst_float> &preds, const MetaInfo &info, int /*iter*/,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels.Size())
        << "labels are not correctly provided"
        << "preds.size=" << preds.Size()
        << ", label.size=" << info.labels.Size();

    const size_t ndata = preds.Size();
    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }
    out_gpair->Resize(ndata);
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx] * 2.0 - 1.0;
          bst_float g, h;
          if (p * y < 1.0) {
            g = -y * w;
            h = w;
          } else {
            g = 0.0;
            h = std::numeric_limits<bst_float>::min();
          }
          _out_gpair[_idx] = GradientPair(g, h);
        },
        common::Range{0, static_cast<int64_t>(ndata)}, this->ctx_->Threads(),
        ctx_->gpu_id).Eval(
            out_gpair, &preds, info.labels.Data(), &info.weights_);
  }

  void PredTransform(HostDeviceVector<bst_float> *io_preds) const override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = _preds[_idx] > 0.0 ? 1.0 : 0.0;
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size()), 1}, this->ctx_->Threads(),
        io_preds->DeviceIdx())
        .Eval(io_preds);
  }

  const char* DefaultEvalMetric() const override {
    return "error";
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("binary:hinge");
  }
  void LoadConfig(Json const &) override {}
};

// register the objective functions
XGBOOST_REGISTER_OBJECTIVE(HingeObj, "binary:hinge")
.describe("Hinge loss. Expects labels to be in [0,1f]")
.set_body([]() { return new HingeObj(); });

}  // namespace obj
}  // namespace xgboost
