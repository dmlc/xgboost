#include "hip/hip_runtime.h"
/*!
 * Copyright 2018 by Contributors
 * \file hinge.cc
 * \brief Provides an implementation of the hinge loss function
 * \author Henry Gouk
 */
#include <xgboost/objective.h>
#include "../common/math.h"
#include "../common/transform.h"
#include "../common/common.h"
#include "../common/span.h"
#include "../common/host_device_vector.h"

namespace xgboost {
namespace obj {

#if defined(XGBOOST_USE_CUDA)
DMLC_REGISTRY_FILE_TAG(hinge_obj_gpu);
#endif  // defined(XGBOOST_USE_CUDA)

class HingeObj : public ObjFunction {
 public:
  HingeObj() = default;

  void Configure(
      const std::vector<std::pair<std::string, std::string> > &args) override {}

  void GetGradient(const HostDeviceVector<bst_float> &preds,
                   const MetaInfo &info,
                   int iter,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels_.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels_.Size())
        << "labels are not correctly provided"
        << "preds.size=" << preds.Size()
        << ", label.size=" << info.labels_.Size();

    const bool is_null_weight = info.weights_.Size() == 0;
    const size_t ndata = preds.Size();
    out_gpair->Resize(ndata);
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx] * 2.0 - 1.0;
          bst_float g, h;
          if (p * y < 1.0) {
            g = -y * w;
            h = w;
          } else {
            g = 0.0;
            h = std::numeric_limits<bst_float>::min();
          }
          _out_gpair[_idx] = GradientPair(g, h);
        },
        common::Range{0, static_cast<int64_t>(ndata)},
        GPUSet::All(tparam_->gpu_id, tparam_->n_gpus, ndata)).Eval(
            out_gpair, &preds, &info.labels_, &info.weights_);
  }

  void PredTransform(HostDeviceVector<bst_float> *io_preds) override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = _preds[_idx] > 0.0 ? 1.0 : 0.0;
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size()), 1},
        GPUSet::All(tparam_->gpu_id, tparam_->n_gpus, io_preds->Size()))
        .Eval(io_preds);
  }

  const char* DefaultEvalMetric() const override {
    return "error";
  }
};

// register the objective functions
XGBOOST_REGISTER_OBJECTIVE(HingeObj, "binary:hinge")
.describe("Hinge loss. Expects labels to be in [0,1f]")
.set_body([]() { return new HingeObj(); });

}  // namespace obj
}  // namespace xgboost
