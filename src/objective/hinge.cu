#include "hip/hip_runtime.h"
/*!
 * Copyright 2018 by Contributors
 * \file hinge.cc
 * \brief Provides an implementation of the hinge loss function
 * \author Henry Gouk
 */
#include <xgboost/objective.h>
#include "../common/math.h"
#include "../common/transform.h"
#include "../common/common.h"
#include "../common/span.h"
#include "../common/host_device_vector.h"

namespace xgboost {
namespace obj {

#if defined(XGBOOST_USE_CUDA)
DMLC_REGISTRY_FILE_TAG(hinge_obj_gpu);
#endif

struct HingeObjParam : public dmlc::Parameter<HingeObjParam> {
  int n_gpus;
  int gpu_id;
  DMLC_DECLARE_PARAMETER(HingeObjParam) {
    DMLC_DECLARE_FIELD(n_gpus).set_default(1).set_lower_bound(-1)
        .describe("Number of GPUs to use for multi-gpu algorithms.");
    DMLC_DECLARE_FIELD(gpu_id)
        .set_lower_bound(0)
        .set_default(0)
        .describe("gpu to use for objective function evaluation");
  }
};

class HingeObj : public ObjFunction {
 public:
  HingeObj() = default;

  void Configure(
      const std::vector<std::pair<std::string, std::string> > &args) override {
    param_.InitAllowUnknown(args);
    devices_ = GPUSet::All(param_.n_gpus).Normalised(param_.gpu_id);
    label_correct_.Resize(devices_.IsEmpty() ? 1 : devices_.Size());
  }

  void GetGradient(const HostDeviceVector<bst_float> &preds,
                   const MetaInfo &info,
                   int iter,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels_.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels_.Size())
        << "labels are not correctly provided"
        << "preds.size=" << preds.Size()
        << ", label.size=" << info.labels_.Size();

    const bool is_null_weight = info.weights_.Size() == 0;
    const size_t ndata = preds.Size();
    out_gpair->Resize(ndata);
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx] * 2.0 - 1.0;
          bst_float g, h;
          if (p * y < 1.0) {
            g = -y * w;
            h = w;
          } else {
            g = 0.0;
            h = std::numeric_limits<bst_float>::min();
          }
          _out_gpair[_idx] = GradientPair(g, h);
        },
        common::Range{0, static_cast<int64_t>(ndata)}, devices_).Eval(
            &label_correct_, out_gpair, &preds, &info.labels_, &info.weights_);
  }

  void PredTransform(HostDeviceVector<bst_float> *io_preds) override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = _preds[_idx] > 0.0 ? 1.0 : 0.0;
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size()), 1}, devices_)
        .Eval(io_preds);
  }

  const char* DefaultEvalMetric() const override {
    return "error";
  }

 private:
  GPUSet devices_;
  HostDeviceVector<int> label_correct_;
  HingeObjParam param_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(HingeObjParam);
// register the objective functions
XGBOOST_REGISTER_OBJECTIVE(HingeObj, "binary:hinge")
.describe("Hinge loss. Expects labels to be in [0,1f]")
.set_body([]() { return new HingeObj(); });

}  // namespace obj
}  // namespace xgboost
