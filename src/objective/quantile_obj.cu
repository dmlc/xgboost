#include "hip/hip_runtime.h"
/**
 * Copyright 2023 by XGBoost contributors
 */
#include <array>                            // std::array
#include <cstddef>                          // std::size_t
#include <cstdint>                          // std::int32_t
#include <vector>                           // std::vector

#include "../common/linalg_op.h"            // ElementWiseKernel,cbegin,cend
#include "../common/quantile_loss_utils.h"  // QuantileLossParam
#include "../common/stats.h"                // Quantile,WeightedQuantile
#include "adaptive.h"                       // UpdateTreeLeaf
#include "dmlc/parameter.h"                 // DMLC_DECLARE_PARAMETER
#include "init_estimation.h"                // CheckInitInputs
#include "xgboost/base.h"                   // GradientPair,XGBOOST_DEVICE,bst_target_t
#include "xgboost/data.h"                   // MetaInfo
#include "xgboost/host_device_vector.h"     // HostDeviceVector
#include "xgboost/json.h"                   // Json,String,ToJson,FromJson
#include "xgboost/linalg.h"                 // Tensor,MakeTensorView,MakeVec
#include "xgboost/objective.h"              // ObjFunction
#include "xgboost/parameter.h"              // XGBoostParameter

#if defined(XGBOOST_USE_CUDA)

#include "../common/linalg_op.cuh"  // ElementWiseKernel
#include "../common/stats.cuh"      // SegmentedQuantile

#endif                              // defined(XGBOOST_USE_CUDA)

namespace xgboost {
namespace obj {
class QuantileRegression : public ObjFunction {
  common::QuantileLossParam param_;
  HostDeviceVector<float> alpha_;

  bst_target_t Targets(MetaInfo const& info) const override {
    auto const& alpha = param_.quantile_alpha.Get();
    CHECK_EQ(alpha.size(), alpha_.Size()) << "The objective is not yet configured.";
    if (info.ShouldHaveLabels()) {
      CHECK_EQ(info.labels.Shape(1), 1)
          << "Multi-target is not yet supported by the quantile loss.";
    }
    CHECK(!alpha.empty());
    // We have some placeholders for multi-target in the quantile loss. But it's not
    // supported as the gbtree doesn't know how to slice the gradient and there's no 3-dim
    // model shape in general.
    auto n_y = std::max(static_cast<std::size_t>(1), info.labels.Shape(1));
    return alpha_.Size() * n_y;
  }

 public:
  void GetGradient(HostDeviceVector<float> const& preds, const MetaInfo& info, std::int32_t iter,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    if (iter == 0) {
      CheckInitInputs(info);
    }
    CHECK_EQ(param_.quantile_alpha.Get().size(), alpha_.Size());

    using SizeT = decltype(info.num_row_);
    SizeT n_targets = this->Targets(info);
    SizeT n_alphas = alpha_.Size();
    CHECK_NE(n_alphas, 0);
    CHECK_GE(n_targets, n_alphas);
    CHECK_EQ(preds.Size(), info.num_row_ * n_targets);

    auto labels = info.labels.View(ctx_->gpu_id);

    out_gpair->SetDevice(ctx_->gpu_id);
    out_gpair->Resize(n_targets * info.num_row_);
    auto gpair =
        linalg::MakeTensorView(ctx_, out_gpair, info.num_row_, n_alphas, n_targets / n_alphas);

    info.weights_.SetDevice(ctx_->gpu_id);
    common::OptionalWeights weight{ctx_->IsCPU() ? info.weights_.ConstHostSpan()
                                                 : info.weights_.ConstDeviceSpan()};

    preds.SetDevice(ctx_->gpu_id);
    auto predt = linalg::MakeVec(&preds);
    auto n_samples = info.num_row_;

    alpha_.SetDevice(ctx_->gpu_id);
    auto alpha = ctx_->IsCPU() ? alpha_.ConstHostSpan() : alpha_.ConstDeviceSpan();

    linalg::ElementWiseKernel(
        ctx_, gpair, [=] XGBOOST_DEVICE(std::size_t i, GradientPair const&) mutable {
          auto [sample_id, quantile_id, target_id] =
              linalg::UnravelIndex(i, n_samples, alpha.size(), n_targets / alpha.size());

          auto d = predt(i) - labels(sample_id, target_id);
          auto h = weight[sample_id];
          if (d >= 0) {
            auto g = (1.0f - alpha[quantile_id]) * weight[sample_id];
            gpair(sample_id, quantile_id, target_id) = GradientPair{g, h};
          } else {
            auto g = (-alpha[quantile_id] * weight[sample_id]);
            gpair(sample_id, quantile_id, target_id) = GradientPair{g, h};
          }
        });
  }

  void InitEstimation(MetaInfo const& info, linalg::Vector<float>* base_score) const override {
    CHECK(!alpha_.Empty());

    auto n_targets = this->Targets(info);
    base_score->SetDevice(ctx_->gpu_id);
    base_score->Reshape(n_targets);

    double sw{0};
    if (ctx_->IsCPU()) {
      auto quantiles = base_score->HostView();
      auto h_weights = info.weights_.ConstHostVector();
      if (info.weights_.Empty()) {
        sw = info.num_row_;
      } else {
        sw = std::accumulate(std::cbegin(h_weights), std::cend(h_weights), 0.0);
      }
      for (bst_target_t t{0}; t < n_targets; ++t) {
        auto alpha = param_.quantile_alpha[t];
        auto h_labels = info.labels.HostView();
        if (h_weights.empty()) {
          quantiles(t) =
              common::Quantile(ctx_, alpha, linalg::cbegin(h_labels), linalg::cend(h_labels));
        } else {
          CHECK_EQ(h_weights.size(), h_labels.Size());
          quantiles(t) = common::WeightedQuantile(ctx_, alpha, linalg::cbegin(h_labels),
                                                  linalg::cend(h_labels), std::cbegin(h_weights));
        }
      }
    } else {
#if defined(XGBOOST_USE_CUDA)
      alpha_.SetDevice(ctx_->gpu_id);
      auto d_alpha = alpha_.ConstDeviceSpan();
      auto d_labels = info.labels.View(ctx_->gpu_id);
      auto seg_it = dh::MakeTransformIterator<std::size_t>(
          thrust::make_counting_iterator(0ul),
          [=] XGBOOST_DEVICE(std::size_t i) { return i * d_labels.Shape(0); });
      CHECK_EQ(d_labels.Shape(1), 1);
      auto val_it = dh::MakeTransformIterator<float>(thrust::make_counting_iterator(0ul),
                                                     [=] XGBOOST_DEVICE(std::size_t i) {
                                                       auto sample_idx = i % d_labels.Shape(0);
                                                       return d_labels(sample_idx, 0);
                                                     });
      auto n = d_labels.Size() * d_alpha.size();
      CHECK_EQ(base_score->Size(), d_alpha.size());
      if (info.weights_.Empty()) {
        common::SegmentedQuantile(ctx_, d_alpha.data(), seg_it, seg_it + d_alpha.size() + 1, val_it,
                                  val_it + n, base_score->Data());
        sw = info.num_row_;
      } else {
        info.weights_.SetDevice(ctx_->gpu_id);
        auto d_weights = info.weights_.ConstDeviceSpan();
        auto weight_it = dh::MakeTransformIterator<float>(thrust::make_counting_iterator(0ul),
                                                          [=] XGBOOST_DEVICE(std::size_t i) {
                                                            auto sample_idx = i % d_labels.Shape(0);
                                                            return d_weights[sample_idx];
                                                          });
        common::SegmentedWeightedQuantile(ctx_, d_alpha.data(), seg_it, seg_it + d_alpha.size() + 1,
                                          val_it, val_it + n, weight_it, weight_it + n,
                                          base_score->Data());
        sw = dh::Reduce(ctx_->CUDACtx()->CTP(), dh::tcbegin(d_weights), dh::tcend(d_weights), 0.0,
                        thrust::plus<double>{});
      }
#else
      common::AssertGPUSupport();
#endif  // defined(XGBOOST_USE_CUDA)
    }

    // For multiple quantiles, we should extend the base score to a vector instead of
    // computing the average. For now, this is a workaround.
    linalg::Vector<float> temp;
    common::Mean(ctx_, *base_score, &temp);
    double meanq = temp(0) * sw;

    std::array<double, 2> dat{meanq, sw};
    collective::GlobalSum(info, &dat);
    std::tie(meanq, sw) = std::tuple_cat(dat);
    meanq /= (sw + kRtEps);
    base_score->Reshape(1);
    base_score->Data()->Fill(meanq);
  }

  void UpdateTreeLeaf(HostDeviceVector<bst_node_t> const& position, MetaInfo const& info,
                      float learning_rate, HostDeviceVector<float> const& prediction,
                      std::int32_t group_idx, RegTree* p_tree) const override {
    auto alpha = param_.quantile_alpha[group_idx];
    ::xgboost::obj::UpdateTreeLeaf(ctx_, position, group_idx, info, learning_rate, prediction,
                                   alpha, p_tree);
  }

  void Configure(Args const& args) override {
    param_.UpdateAllowUnknown(args);
    param_.Validate();
    this->alpha_.HostVector() = param_.quantile_alpha.Get();
  }
  ObjInfo Task() const override { return {ObjInfo::kRegression, true, true}; }
  static char const* Name() { return "reg:quantileerror"; }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String(Name());
    out["quantile_loss_param"] = ToJson(param_);
  }
  void LoadConfig(Json const& in) override {
    CHECK_EQ(get<String const>(in["name"]), Name());
    FromJson(in["quantile_loss_param"], &param_);
    alpha_.HostVector() = param_.quantile_alpha.Get();
  }

  const char* DefaultEvalMetric() const override { return "quantile"; }
  Json DefaultMetricConfig() const override {
    CHECK(param_.GetInitialised());
    Json config{Object{}};
    config["name"] = String{this->DefaultEvalMetric()};
    config["quantile_loss_param"] = ToJson(param_);
    return config;
  }
};

XGBOOST_REGISTER_OBJECTIVE(QuantileRegression, QuantileRegression::Name())
    .describe("Regression with quantile loss.")
    .set_body([]() { return new QuantileRegression(); });

#if defined(XGBOOST_USE_CUDA)
DMLC_REGISTRY_FILE_TAG(quantile_obj_gpu);
#endif  // defined(XGBOOST_USE_CUDA)
}  // namespace obj
}  // namespace xgboost
