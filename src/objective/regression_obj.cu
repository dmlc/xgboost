#include "hip/hip_runtime.h"
/*!
 * Copyright 2015-2019 by Contributors
 * \file regression_obj.cu
 * \brief Definition of single-value regression and classification objectives.
 * \author Tianqi Chen, Kailong Chen
 */

#include <dmlc/omp.h>
#include <xgboost/logging.h>
#include <xgboost/objective.h>
#include <cmath>
#include <memory>
#include <vector>

#include "xgboost/host_device_vector.h"
#include "xgboost/json.h"
#include "xgboost/parameter.h"
#include "xgboost/span.h"

#include "../common/transform.h"
#include "../common/common.h"
#include "./regression_loss.h"


namespace xgboost {
namespace obj {

#if defined(XGBOOST_USE_CUDA)
DMLC_REGISTRY_FILE_TAG(regression_obj_gpu);
#endif  // defined(XGBOOST_USE_CUDA)

struct RegLossParam : public XGBoostParameter<RegLossParam> {
  float scale_pos_weight;
  // declare parameters
  DMLC_DECLARE_PARAMETER(RegLossParam) {
    DMLC_DECLARE_FIELD(scale_pos_weight).set_default(1.0f).set_lower_bound(0.0f)
      .describe("Scale the weight of positive examples by this factor");
  }
};

template<typename Loss>
class RegLossObj : public ObjFunction {
 protected:
  HostDeviceVector<int> label_correct_;

 public:
  RegLossObj() = default;

  void Configure(const std::vector<std::pair<std::string, std::string> >& args) override {
    param_.UpdateAllowUnknown(args);
  }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo &info,
                   int iter,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    if (info.labels_.Size() == 0U) {
      LOG(WARNING) << "Label set is empty.";
    }
    CHECK_EQ(preds.Size(), info.labels_.Size())
        << "labels are not correctly provided"
        << "preds.size=" << preds.Size() << ", label.size=" << info.labels_.Size();
    size_t const ndata = preds.Size();
    out_gpair->Resize(ndata);
    auto device = tparam_->gpu_id;
    label_correct_.Resize(1);
    label_correct_.Fill(1);

    bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }
    auto scale_pos_weight = param_.scale_pos_weight;
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = Loss::PredTransform(_preds[_idx]);
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float label = _labels[_idx];
          if (label == 1.0f) {
            w *= scale_pos_weight;
          }
          if (!Loss::CheckLabel(label)) {
            // If there is an incorrect label, the host code will know.
            _label_correct[0] = 0;
          }
          _out_gpair[_idx] = GradientPair(Loss::FirstOrderGradient(p, label) * w,
                                          Loss::SecondOrderGradient(p, label) * w);
        },
        common::Range{0, static_cast<int64_t>(ndata)}, device).Eval(
            &label_correct_, out_gpair, &preds, &info.labels_, &info.weights_);

    // copy "label correct" flags back to host
    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag == 0) {
        LOG(FATAL) << Loss::LabelErrorMsg();
      }
    }
  }

 public:
  const char* DefaultEvalMetric() const override {
    return Loss::DefaultEvalMetric();
  }

  void PredTransform(HostDeviceVector<float> *io_preds) override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<float> _preds) {
          _preds[_idx] = Loss::PredTransform(_preds[_idx]);
        }, common::Range{0, static_cast<int64_t>(io_preds->Size())},
        tparam_->gpu_id)
        .Eval(io_preds);
  }

  float ProbToMargin(float base_score) const override {
    return Loss::ProbToMargin(base_score);
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String(Loss::Name());
    out["reg_loss_param"] = toJson(param_);
  }

  void LoadConfig(Json const& in) override {
    fromJson(in["reg_loss_param"], &param_);
  }

 protected:
  RegLossParam param_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(RegLossParam);

XGBOOST_REGISTER_OBJECTIVE(SquaredLossRegression, LinearSquareLoss::Name())
.describe("Regression with squared error.")
.set_body([]() { return new RegLossObj<LinearSquareLoss>(); });

XGBOOST_REGISTER_OBJECTIVE(SquareLogError, SquaredLogError::Name())
.describe("Regression with root mean squared logarithmic error.")
.set_body([]() { return new RegLossObj<SquaredLogError>(); });

XGBOOST_REGISTER_OBJECTIVE(LogisticRegression, LogisticRegression::Name())
.describe("Logistic regression for probability regression task.")
.set_body([]() { return new RegLossObj<LogisticRegression>(); });

XGBOOST_REGISTER_OBJECTIVE(LogisticClassification, LogisticClassification::Name())
.describe("Logistic regression for binary classification task.")
.set_body([]() { return new RegLossObj<LogisticClassification>(); });

XGBOOST_REGISTER_OBJECTIVE(LogisticRaw, LogisticRaw::Name())
.describe("Logistic regression for classification, output score "
          "before logistic transformation.")
.set_body([]() { return new RegLossObj<LogisticRaw>(); });

// Deprecated functions
XGBOOST_REGISTER_OBJECTIVE(LinearRegression, "reg:linear")
.describe("Regression with squared error.")
.set_body([]() {
    LOG(WARNING) << "reg:linear is now deprecated in favor of reg:squarederror.";
    return new RegLossObj<LinearSquareLoss>(); });
// End deprecated

// declare parameter
struct PoissonRegressionParam : public XGBoostParameter<PoissonRegressionParam> {
  float max_delta_step;
  DMLC_DECLARE_PARAMETER(PoissonRegressionParam) {
    DMLC_DECLARE_FIELD(max_delta_step).set_lower_bound(0.0f).set_default(0.7f)
        .describe("Maximum delta step we allow each weight estimation to be." \
                  " This parameter is required for possion regression.");
  }
};

// poisson regression for count
class PoissonRegression : public ObjFunction {
 public:
  // declare functions
  void Configure(const std::vector<std::pair<std::string, std::string> >& args) override {
    param_.UpdateAllowUnknown(args);
  }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo &info,
                   int iter,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels_.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels_.Size()) << "labels are not correctly provided";
    size_t const ndata = preds.Size();
    out_gpair->Resize(ndata);
    auto device = tparam_->gpu_id;
    label_correct_.Resize(1);
    label_correct_.Fill(1);

    bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }
    bst_float max_delta_step = param_.max_delta_step;
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx];
          if (y < 0.0f) {
            _label_correct[0] = 0;
          }
          _out_gpair[_idx] = GradientPair{(expf(p) - y) * w,
                                          expf(p + max_delta_step) * w};
        },
        common::Range{0, static_cast<int64_t>(ndata)}, device).Eval(
            &label_correct_, out_gpair, &preds, &info.labels_, &info.weights_);
    // copy "label correct" flags back to host
    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag == 0) {
        LOG(FATAL) << "PoissonRegression: label must be nonnegative";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = expf(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())},
        tparam_->gpu_id)
        .Eval(io_preds);
  }
  void EvalTransform(HostDeviceVector<bst_float> *io_preds) override {
    PredTransform(io_preds);
  }
  bst_float ProbToMargin(bst_float base_score) const override {
    return std::log(base_score);
  }
  const char* DefaultEvalMetric() const override {
    return "poisson-nloglik";
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("count:poisson");
    out["poisson_regression_param"] = toJson(param_);
  }

  void LoadConfig(Json const& in) override {
    fromJson(in["poisson_regression_param"], &param_);
  }

 private:
  PoissonRegressionParam param_;
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(PoissonRegressionParam);

XGBOOST_REGISTER_OBJECTIVE(PoissonRegression, "count:poisson")
.describe("Possion regression for count data.")
.set_body([]() { return new PoissonRegression(); });


// cox regression for survival data (negative values mean they are censored)
class CoxRegression : public ObjFunction {
 public:
  void Configure(
      const std::vector<std::pair<std::string, std::string> > &args) override {}

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo &info,
                   int iter,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels_.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels_.Size()) << "labels are not correctly provided";
    const auto& preds_h = preds.HostVector();
    out_gpair->Resize(preds_h.size());
    auto& gpair = out_gpair->HostVector();
    const std::vector<size_t> &label_order = info.LabelAbsSort();

    const omp_ulong ndata = static_cast<omp_ulong>(preds_h.size()); // NOLINT(*)
    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }

    // pre-compute a sum
    double exp_p_sum = 0;  // we use double because we might need the precision with large datasets
    for (omp_ulong i = 0; i < ndata; ++i) {
      exp_p_sum += std::exp(preds_h[label_order[i]]);
    }

    // start calculating grad and hess
    const auto& labels = info.labels_.HostVector();
    double r_k = 0;
    double s_k = 0;
    double last_exp_p = 0.0;
    double last_abs_y = 0.0;
    double accumulated_sum = 0;
    for (omp_ulong i = 0; i < ndata; ++i) { // NOLINT(*)
      const size_t ind = label_order[i];
      const double p = preds_h[ind];
      const double exp_p = std::exp(p);
      const double w = info.GetWeight(ind);
      const double y = labels[ind];
      const double abs_y = std::abs(y);

      // only update the denominator after we move forward in time (labels are sorted)
      // this is Breslow's method for ties
      accumulated_sum += last_exp_p;
      if (last_abs_y < abs_y) {
        exp_p_sum -= accumulated_sum;
        accumulated_sum = 0;
      } else {
        CHECK(last_abs_y <= abs_y) << "CoxRegression: labels must be in sorted order, " <<
                                      "MetaInfo::LabelArgsort failed!";
      }

      if (y > 0) {
        r_k += 1.0/exp_p_sum;
        s_k += 1.0/(exp_p_sum*exp_p_sum);
      }

      const double grad = exp_p*r_k - static_cast<bst_float>(y > 0);
      const double hess = exp_p*r_k - exp_p*exp_p * s_k;
      gpair.at(ind) = GradientPair(grad * w, hess * w);

      last_abs_y = abs_y;
      last_exp_p = exp_p;
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) override {
    std::vector<bst_float> &preds = io_preds->HostVector();
    const long ndata = static_cast<long>(preds.size()); // NOLINT(*)
#pragma omp parallel for schedule(static)
    for (long j = 0; j < ndata; ++j) {  // NOLINT(*)
      preds[j] = std::exp(preds[j]);
    }
  }
  void EvalTransform(HostDeviceVector<bst_float> *io_preds) override {
    PredTransform(io_preds);
  }
  bst_float ProbToMargin(bst_float base_score) const override {
    return std::log(base_score);
  }
  const char* DefaultEvalMetric() const override {
    return "cox-nloglik";
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("survival:cox");
  }
  void LoadConfig(Json const&) override {}
};

// register the objective function
XGBOOST_REGISTER_OBJECTIVE(CoxRegression, "survival:cox")
.describe("Cox regression for censored survival data (negative labels are considered censored).")
.set_body([]() { return new CoxRegression(); });

// gamma regression
class GammaRegression : public ObjFunction {
 public:
  void Configure(
      const std::vector<std::pair<std::string, std::string> > &args) override {}

  void GetGradient(const HostDeviceVector<bst_float> &preds,
                   const MetaInfo &info,
                   int iter,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels_.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels_.Size()) << "labels are not correctly provided";
    const size_t ndata = preds.Size();
    auto device = tparam_->gpu_id;
    out_gpair->Resize(ndata);
    label_correct_.Resize(1);
    label_correct_.Fill(1);

    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx];
          if (y < 0.0f) {
            _label_correct[0] = 0;
          }
          _out_gpair[_idx] = GradientPair((1 - y / expf(p)) * w, y / expf(p) * w);
        },
        common::Range{0, static_cast<int64_t>(ndata)}, device).Eval(
            &label_correct_, out_gpair, &preds, &info.labels_, &info.weights_);

    // copy "label correct" flags back to host
    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag == 0) {
        LOG(FATAL) << "GammaRegression: label must be nonnegative";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = expf(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())},
        tparam_->gpu_id)
        .Eval(io_preds);
  }
  void EvalTransform(HostDeviceVector<bst_float> *io_preds) override {
    PredTransform(io_preds);
  }
  bst_float ProbToMargin(bst_float base_score) const override {
    return std::log(base_score);
  }
  const char* DefaultEvalMetric() const override {
    return "gamma-nloglik";
  }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("reg:gamma");
  }
  void LoadConfig(Json const&) override {}

 private:
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
XGBOOST_REGISTER_OBJECTIVE(GammaRegression, "reg:gamma")
.describe("Gamma regression for severity data.")
.set_body([]() { return new GammaRegression(); });


// declare parameter
struct TweedieRegressionParam : public XGBoostParameter<TweedieRegressionParam> {
  float tweedie_variance_power;
  DMLC_DECLARE_PARAMETER(TweedieRegressionParam) {
    DMLC_DECLARE_FIELD(tweedie_variance_power).set_range(1.0f, 2.0f).set_default(1.5f)
      .describe("Tweedie variance power.  Must be between in range [1, 2).");
  }
};

// tweedie regression
class TweedieRegression : public ObjFunction {
 public:
  // declare functions
  void Configure(const std::vector<std::pair<std::string, std::string> >& args) override {
    param_.UpdateAllowUnknown(args);
    std::ostringstream os;
    os << "tweedie-nloglik@" << param_.tweedie_variance_power;
    metric_ = os.str();
  }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo &info,
                   int iter,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels_.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels_.Size()) << "labels are not correctly provided";
    const size_t ndata = preds.Size();
    out_gpair->Resize(ndata);

    auto device = tparam_->gpu_id;
    label_correct_.Resize(1);
    label_correct_.Fill(1);

    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }

    const float rho = param_.tweedie_variance_power;
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx];
          if (y < 0.0f) {
            _label_correct[0] = 0;
          }
          bst_float grad = -y * expf((1 - rho) * p) + expf((2 - rho) * p);
          bst_float hess =
              -y * (1 - rho) * \
              std::exp((1 - rho) * p) + (2 - rho) * expf((2 - rho) * p);
          _out_gpair[_idx] = GradientPair(grad * w, hess * w);
        },
        common::Range{0, static_cast<int64_t>(ndata), 1}, device)
        .Eval(&label_correct_, out_gpair, &preds, &info.labels_, &info.weights_);

    // copy "label correct" flags back to host
    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag == 0) {
        LOG(FATAL) << "TweedieRegression: label must be nonnegative";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = expf(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())},
        tparam_->gpu_id)
        .Eval(io_preds);
  }

  bst_float ProbToMargin(bst_float base_score) const override {
    return std::log(base_score);
  }

  const char* DefaultEvalMetric() const override {
    return metric_.c_str();
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("reg:tweedie");
    out["tweedie_regression_param"] = toJson(param_);
  }
  void LoadConfig(Json const& in) override {
    fromJson(in["tweedie_regression_param"], &param_);
  }

 private:
  std::string metric_;
  TweedieRegressionParam param_;
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(TweedieRegressionParam);

XGBOOST_REGISTER_OBJECTIVE(TweedieRegression, "reg:tweedie")
.describe("Tweedie regression for insurance data.")
.set_body([]() { return new TweedieRegression(); });

}  // namespace obj
}  // namespace xgboost
