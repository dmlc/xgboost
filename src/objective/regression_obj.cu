#include "hip/hip_runtime.h"
/*!
 * Copyright 2015-2022 by XGBoost Contributors
 * \file regression_obj.cu
 * \brief Definition of single-value regression and classification objectives.
 * \author Tianqi Chen, Kailong Chen
 */

#include <dmlc/omp.h>
#include <xgboost/logging.h>
#include <xgboost/objective.h>
#include <xgboost/tree_model.h>

#include <cmath>
#include <cstddef>
#include <memory>
#include <vector>

#include "../common/common.h"
#include "../common/linalg_op.h"
#include "../common/pseudo_huber.h"
#include "../common/stats.h"
#include "../common/threading_utils.h"
#include "../common/transform.h"
#include "./regression_loss.h"
#include "xgboost/base.h"
#include "xgboost/data.h"
#include "xgboost/generic_parameters.h"
#include "xgboost/host_device_vector.h"
#include "xgboost/json.h"
#include "xgboost/linalg.h"
#include "xgboost/parameter.h"
#include "xgboost/span.h"

#if defined(XGBOOST_USE_CUDA)
#include "../common/linalg_op.cuh"
#endif  // defined(XGBOOST_USE_CUDA)

namespace xgboost {
namespace obj {
namespace {
void CheckRegInputs(MetaInfo const& info, HostDeviceVector<bst_float> const& preds) {
  CHECK_EQ(info.labels.Shape(0), info.num_row_) << "Invalid shape of labels.";
  CHECK_EQ(info.labels.Size(), preds.Size()) << "Invalid shape of labels.";
  if (!info.weights_.Empty()) {
    CHECK_EQ(info.weights_.Size(), info.num_row_)
        << "Number of weights should be equal to number of data points.";
  }
}
}  // anonymous namespace

#if defined(XGBOOST_USE_CUDA)
DMLC_REGISTRY_FILE_TAG(regression_obj_gpu);
#endif  // defined(XGBOOST_USE_CUDA)

struct RegLossParam : public XGBoostParameter<RegLossParam> {
  float scale_pos_weight;
  // declare parameters
  DMLC_DECLARE_PARAMETER(RegLossParam) {
    DMLC_DECLARE_FIELD(scale_pos_weight).set_default(1.0f).set_lower_bound(0.0f)
      .describe("Scale the weight of positive examples by this factor");
  }
};

template<typename Loss>
class RegLossObj : public ObjFunction {
 protected:
  HostDeviceVector<float> additional_input_;

 public:
  // 0 - label_correct flag, 1 - scale_pos_weight, 2 - is_null_weight
  RegLossObj(): additional_input_(3) {}

  void Configure(const std::vector<std::pair<std::string, std::string> >& args) override {
    param_.UpdateAllowUnknown(args);
  }

  ObjInfo Task() const override { return Loss::Info(); }

  uint32_t Targets(MetaInfo const& info) const override {
    // Multi-target regression.
    return std::max(static_cast<size_t>(1), info.labels.Shape(1));
  }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo &info, int,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    CheckRegInputs(info, preds);
    size_t const ndata = preds.Size();
    out_gpair->Resize(ndata);
    auto device = ctx_->gpu_id;
    additional_input_.HostVector().begin()[0] = 1;  // Fill the label_correct flag

    bool is_null_weight = info.weights_.Size() == 0;
    auto scale_pos_weight = param_.scale_pos_weight;
    additional_input_.HostVector().begin()[1] = scale_pos_weight;
    additional_input_.HostVector().begin()[2] = is_null_weight;

    const size_t nthreads = ctx_->Threads();
    bool on_device = device >= 0;
    // On CPU we run the transformation each thread processing a contigious block of data
    // for better performance.
    const size_t n_data_blocks = std::max(static_cast<size_t>(1), (on_device ? ndata : nthreads));
    const size_t block_size = ndata / n_data_blocks + !!(ndata % n_data_blocks);
    auto const n_targets = std::max(info.labels.Shape(1), static_cast<size_t>(1));

    common::Transform<>::Init(
        [block_size, ndata, n_targets] XGBOOST_DEVICE(
            size_t data_block_idx, common::Span<float> _additional_input,
            common::Span<GradientPair> _out_gpair,
            common::Span<const bst_float> _preds,
            common::Span<const bst_float> _labels,
            common::Span<const bst_float> _weights) {
          const bst_float* preds_ptr = _preds.data();
          const bst_float* labels_ptr = _labels.data();
          const bst_float* weights_ptr = _weights.data();
          GradientPair* out_gpair_ptr = _out_gpair.data();
          const size_t begin = data_block_idx*block_size;
          const size_t end = std::min(ndata, begin + block_size);
          const float _scale_pos_weight = _additional_input[1];
          const bool _is_null_weight = _additional_input[2];

          for (size_t idx = begin; idx < end; ++idx) {
            bst_float p = Loss::PredTransform(preds_ptr[idx]);
            bst_float w = _is_null_weight ? 1.0f : weights_ptr[idx / n_targets];
            bst_float label = labels_ptr[idx];
            if (label == 1.0f) {
              w *= _scale_pos_weight;
            }
            if (!Loss::CheckLabel(label)) {
              // If there is an incorrect label, the host code will know.
              _additional_input[0] = 0;
            }
            out_gpair_ptr[idx] = GradientPair(Loss::FirstOrderGradient(p, label) * w,
                                              Loss::SecondOrderGradient(p, label) * w);
          }
        },
        common::Range{0, static_cast<int64_t>(n_data_blocks)}, nthreads, device)
        .Eval(&additional_input_, out_gpair, &preds, info.labels.Data(),
              &info.weights_);

    auto const flag = additional_input_.HostVector().begin()[0];
    if (flag == 0) {
      LOG(FATAL) << Loss::LabelErrorMsg();
    }
  }

 public:
  const char* DefaultEvalMetric() const override {
    return Loss::DefaultEvalMetric();
  }

  void PredTransform(HostDeviceVector<float> *io_preds) const override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<float> _preds) {
          _preds[_idx] = Loss::PredTransform(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())}, this->ctx_->Threads(),
        io_preds->DeviceIdx())
        .Eval(io_preds);
  }

  float ProbToMargin(float base_score) const override {
    return Loss::ProbToMargin(base_score);
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String(Loss::Name());
    out["reg_loss_param"] = ToJson(param_);
  }

  void LoadConfig(Json const& in) override {
    FromJson(in["reg_loss_param"], &param_);
  }

 protected:
  RegLossParam param_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(RegLossParam);

XGBOOST_REGISTER_OBJECTIVE(SquaredLossRegression, LinearSquareLoss::Name())
.describe("Regression with squared error.")
.set_body([]() { return new RegLossObj<LinearSquareLoss>(); });

XGBOOST_REGISTER_OBJECTIVE(SquareLogError, SquaredLogError::Name())
.describe("Regression with root mean squared logarithmic error.")
.set_body([]() { return new RegLossObj<SquaredLogError>(); });

XGBOOST_REGISTER_OBJECTIVE(LogisticRegression, LogisticRegression::Name())
.describe("Logistic regression for probability regression task.")
.set_body([]() { return new RegLossObj<LogisticRegression>(); });

XGBOOST_REGISTER_OBJECTIVE(LogisticClassification, LogisticClassification::Name())
.describe("Logistic regression for binary classification task.")
.set_body([]() { return new RegLossObj<LogisticClassification>(); });

XGBOOST_REGISTER_OBJECTIVE(LogisticRaw, LogisticRaw::Name())
.describe("Logistic regression for classification, output score "
          "before logistic transformation.")
.set_body([]() { return new RegLossObj<LogisticRaw>(); });

// Deprecated functions
XGBOOST_REGISTER_OBJECTIVE(LinearRegression, "reg:linear")
.describe("Regression with squared error.")
.set_body([]() {
    LOG(WARNING) << "reg:linear is now deprecated in favor of reg:squarederror.";
    return new RegLossObj<LinearSquareLoss>(); });
// End deprecated

class PseudoHuberRegression : public ObjFunction {
  PesudoHuberParam param_;

 public:
  void Configure(Args const& args) override { param_.UpdateAllowUnknown(args); }
  ObjInfo Task() const override { return ObjInfo::kRegression; }
  uint32_t Targets(MetaInfo const& info) const override {
    return std::max(static_cast<size_t>(1), info.labels.Shape(1));
  }

  void GetGradient(HostDeviceVector<bst_float> const& preds, const MetaInfo& info, int iter,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    CheckRegInputs(info, preds);
    auto slope = param_.huber_slope;
    CHECK_NE(slope, 0.0) << "slope for pseudo huber cannot be 0.";
    auto labels = info.labels.View(ctx_->gpu_id);

    out_gpair->SetDevice(ctx_->gpu_id);
    out_gpair->Resize(info.labels.Size());
    auto gpair = linalg::MakeVec(out_gpair);

    preds.SetDevice(ctx_->gpu_id);
    auto predt = linalg::MakeVec(&preds);

    info.weights_.SetDevice(ctx_->gpu_id);
    common::OptionalWeights weight{ctx_->IsCPU() ? info.weights_.ConstHostSpan()
                                                 : info.weights_.ConstDeviceSpan()};

    linalg::ElementWiseKernel(ctx_, labels, [=] XGBOOST_DEVICE(size_t i, float const y) mutable {
      auto sample_id = std::get<0>(linalg::UnravelIndex(i, labels.Shape()));
      const float z = predt(i) - y;
      const float scale_sqrt = std::sqrt(1 + common::Sqr(z) / common::Sqr(slope));
      float grad = z / scale_sqrt;

      auto scale = common::Sqr(slope) + common::Sqr(z);
      float hess = common::Sqr(slope) / (scale * scale_sqrt);

      auto w = weight[sample_id];
      gpair(i) = {grad * w, hess * w};
    });
  }

  const char* DefaultEvalMetric() const override { return "mphe"; }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("reg:pseudohubererror");
    out["pseduo_huber_param"] = ToJson(param_);
  }

  void LoadConfig(Json const& in) override {
    auto const& config = get<Object const>(in);
    if (config.find("pseduo_huber_param") == config.cend()) {
      // The parameter is added in 1.6.
      return;
    }
    FromJson(in["pseduo_huber_param"], &param_);
  }
};

XGBOOST_REGISTER_OBJECTIVE(PseudoHuberRegression, "reg:pseudohubererror")
    .describe("Regression Pseudo Huber error.")
    .set_body([]() { return new PseudoHuberRegression(); });

// declare parameter
struct PoissonRegressionParam : public XGBoostParameter<PoissonRegressionParam> {
  float max_delta_step;
  DMLC_DECLARE_PARAMETER(PoissonRegressionParam) {
    DMLC_DECLARE_FIELD(max_delta_step).set_lower_bound(0.0f).set_default(0.7f)
        .describe("Maximum delta step we allow each weight estimation to be." \
                  " This parameter is required for possion regression.");
  }
};

// poisson regression for count
class PoissonRegression : public ObjFunction {
 public:
  // declare functions
  void Configure(const std::vector<std::pair<std::string, std::string> >& args) override {
    param_.UpdateAllowUnknown(args);
  }

  ObjInfo Task() const override { return ObjInfo::kRegression; }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo &info, int,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels.Size()) << "labels are not correctly provided";
    size_t const ndata = preds.Size();
    out_gpair->Resize(ndata);
    auto device = ctx_->gpu_id;
    label_correct_.Resize(1);
    label_correct_.Fill(1);

    bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }
    bst_float max_delta_step = param_.max_delta_step;
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx];
          if (y < 0.0f) {
            _label_correct[0] = 0;
          }
          _out_gpair[_idx] = GradientPair{(expf(p) - y) * w,
                                          expf(p + max_delta_step) * w};
        },
        common::Range{0, static_cast<int64_t>(ndata)}, this->ctx_->Threads(), device).Eval(
            &label_correct_, out_gpair, &preds, info.labels.Data(), &info.weights_);
    // copy "label correct" flags back to host
    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag == 0) {
        LOG(FATAL) << "PoissonRegression: label must be nonnegative";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) const override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = expf(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())}, this->ctx_->Threads(),
        io_preds->DeviceIdx())
        .Eval(io_preds);
  }
  void EvalTransform(HostDeviceVector<bst_float> *io_preds) override {
    PredTransform(io_preds);
  }
  bst_float ProbToMargin(bst_float base_score) const override {
    return std::log(base_score);
  }
  const char* DefaultEvalMetric() const override {
    return "poisson-nloglik";
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("count:poisson");
    out["poisson_regression_param"] = ToJson(param_);
  }

  void LoadConfig(Json const& in) override {
    FromJson(in["poisson_regression_param"], &param_);
  }

 private:
  PoissonRegressionParam param_;
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(PoissonRegressionParam);

XGBOOST_REGISTER_OBJECTIVE(PoissonRegression, "count:poisson")
.describe("Poisson regression for count data.")
.set_body([]() { return new PoissonRegression(); });


// cox regression for survival data (negative values mean they are censored)
class CoxRegression : public ObjFunction {
 public:
  void Configure(Args const&) override {}
  ObjInfo Task() const override { return ObjInfo::kRegression; }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo &info, int,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels.Size()) << "labels are not correctly provided";
    const auto& preds_h = preds.HostVector();
    out_gpair->Resize(preds_h.size());
    auto& gpair = out_gpair->HostVector();
    const std::vector<size_t> &label_order = info.LabelAbsSort();

    const omp_ulong ndata = static_cast<omp_ulong>(preds_h.size()); // NOLINT(*)
    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }

    // pre-compute a sum
    double exp_p_sum = 0;  // we use double because we might need the precision with large datasets
    for (omp_ulong i = 0; i < ndata; ++i) {
      exp_p_sum += std::exp(preds_h[label_order[i]]);
    }

    // start calculating grad and hess
    const auto& labels = info.labels.HostView();
    double r_k = 0;
    double s_k = 0;
    double last_exp_p = 0.0;
    double last_abs_y = 0.0;
    double accumulated_sum = 0;
    for (omp_ulong i = 0; i < ndata; ++i) { // NOLINT(*)
      const size_t ind = label_order[i];
      const double p = preds_h[ind];
      const double exp_p = std::exp(p);
      const double w = info.GetWeight(ind);
      const double y = labels(ind);
      const double abs_y = std::abs(y);

      // only update the denominator after we move forward in time (labels are sorted)
      // this is Breslow's method for ties
      accumulated_sum += last_exp_p;
      if (last_abs_y < abs_y) {
        exp_p_sum -= accumulated_sum;
        accumulated_sum = 0;
      } else {
        CHECK(last_abs_y <= abs_y) << "CoxRegression: labels must be in sorted order, " <<
                                      "MetaInfo::LabelArgsort failed!";
      }

      if (y > 0) {
        r_k += 1.0/exp_p_sum;
        s_k += 1.0/(exp_p_sum*exp_p_sum);
      }

      const double grad = exp_p*r_k - static_cast<bst_float>(y > 0);
      const double hess = exp_p*r_k - exp_p*exp_p * s_k;
      gpair.at(ind) = GradientPair(grad * w, hess * w);

      last_abs_y = abs_y;
      last_exp_p = exp_p;
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) const override {
    std::vector<bst_float> &preds = io_preds->HostVector();
    const long ndata = static_cast<long>(preds.size()); // NOLINT(*)
    common::ParallelFor(ndata, ctx_->Threads(), [&](long j) { // NOLINT(*)
      preds[j] = std::exp(preds[j]);
    });
  }
  void EvalTransform(HostDeviceVector<bst_float> *io_preds) override {
    PredTransform(io_preds);
  }
  bst_float ProbToMargin(bst_float base_score) const override {
    return std::log(base_score);
  }
  const char* DefaultEvalMetric() const override {
    return "cox-nloglik";
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("survival:cox");
  }
  void LoadConfig(Json const&) override {}
};

// register the objective function
XGBOOST_REGISTER_OBJECTIVE(CoxRegression, "survival:cox")
.describe("Cox regression for censored survival data (negative labels are considered censored).")
.set_body([]() { return new CoxRegression(); });

// gamma regression
class GammaRegression : public ObjFunction {
 public:
  void Configure(Args const&) override {}
  ObjInfo Task() const override { return ObjInfo::kRegression; }

  void GetGradient(const HostDeviceVector<bst_float> &preds,
                   const MetaInfo &info, int,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels.Size()) << "labels are not correctly provided";
    const size_t ndata = preds.Size();
    auto device = ctx_->gpu_id;
    out_gpair->Resize(ndata);
    label_correct_.Resize(1);
    label_correct_.Fill(1);

    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx];
          if (y <= 0.0f) {
            _label_correct[0] = 0;
          }
          _out_gpair[_idx] = GradientPair((1 - y / expf(p)) * w, y / expf(p) * w);
        },
        common::Range{0, static_cast<int64_t>(ndata)}, this->ctx_->Threads(), device).Eval(
            &label_correct_, out_gpair, &preds, info.labels.Data(), &info.weights_);

    // copy "label correct" flags back to host
    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag == 0) {
        LOG(FATAL) << "GammaRegression: label must be positive.";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) const override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = expf(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())}, this->ctx_->Threads(),
        io_preds->DeviceIdx())
        .Eval(io_preds);
  }
  void EvalTransform(HostDeviceVector<bst_float> *io_preds) override {
    PredTransform(io_preds);
  }
  bst_float ProbToMargin(bst_float base_score) const override {
    return std::log(base_score);
  }
  const char* DefaultEvalMetric() const override {
    return "gamma-nloglik";
  }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("reg:gamma");
  }
  void LoadConfig(Json const&) override {}

 private:
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
XGBOOST_REGISTER_OBJECTIVE(GammaRegression, "reg:gamma")
.describe("Gamma regression for severity data.")
.set_body([]() { return new GammaRegression(); });


// declare parameter
struct TweedieRegressionParam : public XGBoostParameter<TweedieRegressionParam> {
  float tweedie_variance_power;
  DMLC_DECLARE_PARAMETER(TweedieRegressionParam) {
    DMLC_DECLARE_FIELD(tweedie_variance_power).set_range(1.0f, 2.0f).set_default(1.5f)
      .describe("Tweedie variance power.  Must be between in range [1, 2).");
  }
};

// tweedie regression
class TweedieRegression : public ObjFunction {
 public:
  // declare functions
  void Configure(const std::vector<std::pair<std::string, std::string> >& args) override {
    param_.UpdateAllowUnknown(args);
    std::ostringstream os;
    os << "tweedie-nloglik@" << param_.tweedie_variance_power;
    metric_ = os.str();
  }

  ObjInfo Task() const override { return ObjInfo::kRegression; }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo &info, int,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels.Size()) << "labels are not correctly provided";
    const size_t ndata = preds.Size();
    out_gpair->Resize(ndata);

    auto device = ctx_->gpu_id;
    label_correct_.Resize(1);
    label_correct_.Fill(1);

    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }

    const float rho = param_.tweedie_variance_power;
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx];
          if (y < 0.0f) {
            _label_correct[0] = 0;
          }
          bst_float grad = -y * expf((1 - rho) * p) + expf((2 - rho) * p);
          bst_float hess =
              -y * (1 - rho) * \
              std::exp((1 - rho) * p) + (2 - rho) * expf((2 - rho) * p);
          _out_gpair[_idx] = GradientPair(grad * w, hess * w);
        },
        common::Range{0, static_cast<int64_t>(ndata), 1}, this->ctx_->Threads(), device)
        .Eval(&label_correct_, out_gpair, &preds, info.labels.Data(), &info.weights_);

    // copy "label correct" flags back to host
    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag == 0) {
        LOG(FATAL) << "TweedieRegression: label must be nonnegative";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) const override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = expf(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())}, this->ctx_->Threads(),
        io_preds->DeviceIdx())
        .Eval(io_preds);
  }

  bst_float ProbToMargin(bst_float base_score) const override {
    return std::log(base_score);
  }

  const char* DefaultEvalMetric() const override {
    return metric_.c_str();
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("reg:tweedie");
    out["tweedie_regression_param"] = ToJson(param_);
  }
  void LoadConfig(Json const& in) override {
    FromJson(in["tweedie_regression_param"], &param_);
  }

 private:
  std::string metric_;
  TweedieRegressionParam param_;
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(TweedieRegressionParam);

XGBOOST_REGISTER_OBJECTIVE(TweedieRegression, "reg:tweedie")
.describe("Tweedie regression for insurance data.")
.set_body([]() { return new TweedieRegression(); });

void SegmentedPercentile(Context const* ctx, double alpha, RowIndexCache const& row_index,
                         MetaInfo const& info, HostDeviceVector<float> const& predt,
                         HostDeviceVector<float>* quantiles) {
  CHECK(alpha >= 0 && alpha <= 1);

  auto d_predt = predt.ConstDeviceSpan();
  auto d_labels = info.labels.View(ctx->gpu_id);
  linalg::Tensor<float, 2> residue{d_labels.Shape(), ctx->gpu_id};
  auto d_residue = residue.View(ctx->gpu_id);
  CHECK_EQ(d_predt.size(), d_labels.Size());
  linalg::ElementWiseKernel(ctx, d_labels, [=] XGBOOST_DEVICE(size_t i, float y) mutable {
    size_t sample_id, target_id;
    std::tie(sample_id, target_id) = linalg::UnravelIndex(i, d_labels.Shape());
    d_residue(sample_id, target_id) = y - d_predt[i];
  });

  dh::device_vector<size_t> segment_idx(row_index.indptr.size() + 1, 0);
  auto d_segment_idx = dh::ToSpan(segment_idx);
  dh::device_vector<RowIndexCache::Segment> indptr(row_index.indptr);
  auto d_indptr = dh::ToSpan(indptr);
  dh::LaunchN(d_segment_idx.size(), [=] XGBOOST_DEVICE(size_t i) {
    if (i == d_segment_idx.size() - 1) {
      d_segment_idx[i] = d_indptr[i].begin + d_indptr[i].n;
      return;
    }
    d_segment_idx[i] = d_indptr[i].begin;
  });

  dh::XGBDeviceAllocator<char> alloc;
  dh::device_vector<size_t> sorted_idx(d_labels.Shape(0));
  dh::Iota(dh::ToSpan(sorted_idx));
  using Tup = thrust::tuple<size_t, float>;
  auto key_it = dh::MakeTransformIterator<Tup>(
      thrust::make_counting_iterator(0ul), [=] XGBOOST_DEVICE(size_t i) -> Tup {
        size_t sample_id, target_id;
        std::tie(sample_id, target_id) = linalg::UnravelIndex(i, d_labels.Shape());
        auto leaf_idx = dh::SegmentId(d_segment_idx, sample_id);
        auto residue = d_residue(sample_id, target_id);
        return thrust::make_tuple(leaf_idx, residue);
      });
  dh::device_vector<Tup> keys(residue.Size());
  dh::XGBCachingDeviceAllocator<char> caching;
  thrust::copy(thrust::cuda::par(caching), key_it, key_it + keys.size(), keys.begin());

  thrust::stable_sort_by_key(thrust::cuda::par(alloc), keys.begin(), keys.end(), sorted_idx.begin(),
                             [=] XGBOOST_DEVICE(Tup const& l, Tup const& r) {
                               if (thrust::get<0>(l) != thrust::get<0>(r)) {
                                 return thrust::get<0>(l) < thrust::get<0>(r);  // segment index
                               }
                               return thrust::get<1>(l) < thrust::get<1>(r);  // residue
                             });

  dh::caching_device_vector<RowIndexCache::Segment> segment(row_index.indptr.size());
  thrust::copy(row_index.indptr.cbegin(), row_index.indptr.cend(), segment.begin());
  auto d_segments = dh::ToSpan(segment);

  quantiles->Resize(row_index.indptr.size());
  auto d_results = quantiles->DeviceSpan();

  auto d_row_index = row_index.row_index.ConstDeviceSpan();
  auto d_sorted_idx = dh::ToSpan(sorted_idx);
  auto d_keys = dh::ToSpan(keys);

  dh::LaunchN(residue.Size(), [=] XGBOOST_DEVICE(size_t i) {
    size_t sample_id, target_id;
    std::tie(sample_id, target_id) = linalg::UnravelIndex(i, d_labels.Shape());
    // each segment is the index of a leaf.
    size_t seg_idx = thrust::get<0>(d_keys[i]);
    auto seg = d_segments[seg_idx];

    double x = alpha * static_cast<double>(seg.n + 1);
    double k = std::floor(x) - 1;
    double d = (x - 1) - k;

    if (i == seg.begin) {
      auto v0 = d_residue(d_row_index[d_sorted_idx[static_cast<size_t>(k)]], target_id);
      auto v1 = d_residue(d_row_index[d_sorted_idx[static_cast<size_t>(k) + 1]], target_id);
      d_results[seg_idx] = v0 + d * (v1 - v0);
    }
  });
}

void UpdateTreeLeafDevice(Context const* ctx, common::Span<RowIndexCache const> row_index,
                          MetaInfo const& info, HostDeviceVector<float> const& prediction,
                          uint32_t target, float alpha, RegTree* p_tree) {
  dh::safe_cuda(hipSetDevice(ctx->gpu_id));
  CHECK_EQ(row_index.size(), 1)
      << "External memory with GPU hist should have only 1 row partition.";
  auto const& part = row_index.front();

  HostDeviceVector<float> results;
  SegmentedPercentile(ctx, alpha, part, info, prediction, &results);

  auto const& h_results = results.HostVector();
  auto& tree = *p_tree;
  for (size_t i = 0; i < row_index.front().indptr.size(); ++i) {
    auto seg = row_index.front().indptr[i];
    auto q = h_results[i];
    CHECK(tree[seg.nidx].IsLeaf());
    tree[seg.nidx].SetLeaf(q);  // fixme: exact tree method
  }
}

void UpdateTreeLeafHost(Context const* ctx, common::Span<RowIndexCache const> row_index,
                        MetaInfo const& info, HostDeviceVector<float> const& prediction,
                        uint32_t target, float alpha, RegTree* p_tree) {
  auto& tree = *p_tree;
  std::vector<float> quantiles;
  for (auto const& part : row_index) {
    std::vector<float> results(part.indptr.size());
    common::ParallelFor(part.indptr.size(), ctx->Threads(), [&](size_t k) {
      auto const& seg = part.indptr[k];
      CHECK(tree[seg.nidx].IsLeaf());
      auto h_row_set = part.row_index.HostSpan().subspan(seg.begin, seg.n);
      float q{0};
      auto h_labels = info.labels.HostView().Slice(linalg::All(), target);
      auto const& h_prediction = prediction.ConstHostVector();
      auto iter = common::MakeIndexTransformIter([&](size_t i) -> float {
        auto row_idx = h_row_set[i];
        return h_labels(row_idx) - h_prediction[row_idx];
      });

      if (info.weights_.Empty()) {
        q = common::Percentile(alpha, iter, iter + h_row_set.size());
      } else {
        q = common::WeightedPercentile(alpha, h_row_set,
                                       info.labels.HostView().Slice(linalg::All(), target),
                                       linalg::MakeVec(&info.weights_));
      }
      results.at(k) = q;
    });

    // fixme: verify this is correct for external memory
    if (quantiles.empty()) {
      quantiles.resize(results.size(), 0);
    }
    for (size_t i = 0; i < results.size(); ++i) {
      quantiles[i] += results[i];
    }
  }

  // use the mean value
  rabit::Allreduce<rabit::op::Sum>(quantiles.data(), quantiles.size());
  auto world = rabit::GetWorldSize();
  std::transform(quantiles.begin(), quantiles.end(), quantiles.begin(),
                 [&](float q) { return q / world; });

  // fixme: verify this is correct for external memory
  for (size_t i = 0; i < row_index.front().indptr.size(); ++i) {
    auto seg = row_index.front().indptr[i];
    auto q = quantiles[i];
    CHECK(tree[seg.nidx].IsLeaf());
    tree[seg.nidx].SetLeaf(q);  // fixme: exact tree method
  }
}

class MeanAbsoluteError : public ObjFunction {
 public:
  void Configure(Args const&) override {}

  uint32_t Targets(MetaInfo const& info) const override {
    return std::max(static_cast<size_t>(1), info.labels.Shape(1));
  }

  struct ObjInfo Task() const override {
    return {ObjInfo::kRegression, true, true};
  }

  void GetGradient(HostDeviceVector<bst_float> const& preds, const MetaInfo& info, int iter,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    auto labels = info.labels.View(ctx_->gpu_id);

    out_gpair->SetDevice(ctx_->gpu_id);
    out_gpair->Resize(info.labels.Size());
    auto gpair = linalg::MakeVec(out_gpair);

    preds.SetDevice(ctx_->gpu_id);
    auto predt = linalg::MakeVec(&preds);
    info.weights_.SetDevice(ctx_->gpu_id);
    common::OptionalWeights weight{ctx_->IsCPU() ? info.weights_.ConstHostSpan()
                                                 : info.weights_.ConstDeviceSpan()};

    linalg::ElementWiseKernel(ctx_, labels, [=] XGBOOST_DEVICE(size_t i, float const y) mutable {
      auto sign = [](auto x) {
        return (x > static_cast<decltype(x)>(0)) - (x < static_cast<decltype(x)>(0));
      };
      auto sample_id = std::get<0>(linalg::UnravelIndex(i, labels.Shape()));
      auto grad = sign(predt(i) - y) * weight[i];
      auto hess = weight[sample_id];
      gpair(i) = GradientPair{grad, hess};
    });
  }

  void UpdateTreeLeaf(common::Span<RowIndexCache const> row_index, MetaInfo const& info,
                      HostDeviceVector<float> const& prediction, uint32_t target,
                      RegTree* p_tree) const override {
    if (ctx_->IsCPU()) {
      UpdateTreeLeafHost(ctx_, row_index, info, prediction, target, 0.5, p_tree);
    } else {
      UpdateTreeLeafDevice(ctx_, row_index, info, prediction, target, 0.5, p_tree);
    }
  }

  const char* DefaultEvalMetric() const override { return "mae"; }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("reg:absoluteerror");
  }

  void LoadConfig(Json const& in) override {}
};

XGBOOST_REGISTER_OBJECTIVE(MeanAbsoluteError, "reg:absoluteerror")
    .describe("Mean absoluate error.")
    .set_body([]() { return new MeanAbsoluteError(); });
}  // namespace obj
}  // namespace xgboost
