#include "hip/hip_runtime.h"
/**
 * Copyright 2015-2025, XGBoost Contributors
 * \file regression_obj.cu
 * \brief Definition of single-value regression and classification objectives.
 * \author Tianqi Chen, Kailong Chen
 */
#include <dmlc/omp.h>

#include <algorithm>  // for all_of
#include <cmath>
#include <cstdint>  // for  int32_t
#include <vector>   // for vector

#include "../common/common.h"
#include "../common/linalg_op.h"
#include "../common/numeric.h"          // Reduce
#include "../common/optional_weight.h"  // OptionalWeights
#include "../common/pseudo_huber.h"
#include "../common/stats.h"
#include "../common/threading_utils.h"
#include "../common/transform.h"
#include "../common/utils.h"  // for NoOp
#include "./regression_loss.h"
#include "adaptive.h"
#include "init_estimation.h"  // FitIntercept
#include "regression_param.h"
#include "xgboost/base.h"
#include "xgboost/context.h"  // Context
#include "xgboost/data.h"     // MetaInfo
#include "xgboost/host_device_vector.h"
#include "xgboost/json.h"
#include "xgboost/linalg.h"
#include "xgboost/logging.h"
#include "xgboost/objective.h"  // ObjFunction
#include "xgboost/parameter.h"
#include "xgboost/span.h"
#include "xgboost/tree_model.h"  // RegTree

#if defined(XGBOOST_USE_CUDA)
#include "../common/algorithm.cuh"       // for AllOf
#include "../common/cuda_context.cuh"    // for HIPContext
#include "../common/device_helpers.cuh"  // for MakeIndexTransformIter
#include "../common/linalg_op.cuh"
#endif  // defined(XGBOOST_USE_CUDA)

#if defined(XGBOOST_USE_SYCL)
#include "../../plugin/sycl/common/linalg_op.h"
#endif

namespace xgboost::obj {
namespace {
void CheckRegInputs(MetaInfo const& info, HostDeviceVector<float> const& preds) {
  CheckInitInputs(info);
  CHECK_EQ(info.labels.Size(), preds.Size()) << "Invalid shape of labels.";
}

template <typename Loss>
void ValidateLabel(Context const* ctx, MetaInfo const& info) {
  auto label = info.labels.View(ctx->Device());
  auto valid = ctx->DispatchDevice(
      [&] {
        return std::all_of(linalg::cbegin(label), linalg::cend(label),
                           [](float y) -> bool { return Loss::CheckLabel(y); });
      },
      [&] {
#if defined(XGBOOST_USE_CUDA)
        auto it = dh::MakeIndexTransformIter([=] XGBOOST_DEVICE(std::size_t i) -> float {
          auto [m, n] = linalg::UnravelIndex(i, label.Shape());
          return label(m, n);
        });
        return common::AllOf(ctx->CUDACtx()->CTP(), it, it + label.Size(),
                             [] XGBOOST_DEVICE(float y) { return Loss::CheckLabel(y); });
#else
        common::AssertGPUSupport();
        return false;
#endif  // defined(XGBOOST_USE_CUDA)
      },
      [&] {
#if defined(XGBOOST_USE_SYCL)
        return sycl::linalg::Validate(ctx->Device(), label,
                                      [](float y) -> bool { return Loss::CheckLabel(y); });
#else
        common::AssertSYCLSupport();
        return false;
#endif  // defined(XGBOOST_USE_SYCL)
      });
  if (!valid) {
    LOG(FATAL) << Loss::LabelErrorMsg();
  }
  if (!info.weights_.Empty()) {
    CHECK_EQ(info.weights_.Size(), info.num_row_)
        << "Number of weights should be equal to the number of data points.";
  }
}

template <typename Fn, typename Chk = common::NoOp<bool>, typename Err = common::NoOp<StringView>>
void ProbToMarginImpl(Context const* ctx, linalg::Vector<float>* base_score, Fn&& fn,
                      Chk check = common::NoOp{true}, Err error = common::NoOp<StringView>{{}}) {
  auto intercept = base_score->View(ctx->Device());
  bool is_valid = ctx->DispatchDevice(
      [&] { return std::all_of(linalg::cbegin(intercept), linalg::cend(intercept), check); },
      [&] {
#if defined(XGBOOST_USE_CUDA)
        return common::AllOf(ctx->CUDACtx()->CTP(), linalg::tcbegin(intercept),
                             linalg::tcend(intercept), check);
#else
        common::AssertGPUSupport();
        return false;
#endif  // defined(XGBOOST_USE_CUDA)
      });
  CHECK(is_valid) << error();
  linalg::ElementWiseKernel(ctx, intercept, [=] XGBOOST_DEVICE(std::size_t i) mutable {
    intercept(i) = fn(intercept(i));
  });
}
}  // anonymous namespace

#if defined(XGBOOST_USE_CUDA)
DMLC_REGISTRY_FILE_TAG(regression_obj_gpu);
#endif  // defined(XGBOOST_USE_CUDA)

template<typename Loss>
class RegLossObj : public FitInterceptGlmLike {
 protected:
  HostDeviceVector<float> additional_input_;

 public:
  // 0 - scale_pos_weight, 1 - is_null_weight
  RegLossObj() : additional_input_(2) {}

  void Configure(Args const& args) override { param_.UpdateAllowUnknown(args); }

  [[nodiscard]] ObjInfo Task() const override { return Loss::Info(); }

  [[nodiscard]] bst_target_t Targets(MetaInfo const& info) const override {
    // Multi-target regression.
    return std::max(static_cast<std::size_t>(1), info.labels.Shape(1));
  }

  void GetGradient(const HostDeviceVector<float>& preds, const MetaInfo& info, std::int32_t iter,
                   linalg::Matrix<GradientPair>* out_gpair) override {
    CheckRegInputs(info, preds);
    if (iter == 0) {
      ValidateLabel<Loss>(this->ctx_, info);
    }

    size_t const ndata = preds.Size();
    out_gpair->SetDevice(ctx_->Device());
    auto device = ctx_->Device();

    bool is_null_weight = info.weights_.Size() == 0;
    auto scale_pos_weight = param_.scale_pos_weight;
    additional_input_.HostVector().begin()[0] = scale_pos_weight;
    additional_input_.HostVector().begin()[1] = is_null_weight;

    const size_t nthreads = ctx_->Threads();
    bool on_device = !device.IsCPU();
    // On CPU we run the transformation each thread processing a contigious block of data
    // for better performance.
    const size_t n_data_blocks = std::max(static_cast<size_t>(1), (on_device ? ndata : nthreads));
    const size_t block_size = ndata / n_data_blocks + !!(ndata % n_data_blocks);
    auto const n_targets = this->Targets(info);
    out_gpair->Reshape(info.num_row_, n_targets);

    common::Transform<>::Init(
        [block_size, ndata, n_targets] XGBOOST_DEVICE(
            size_t data_block_idx, common::Span<float> _additional_input,
            common::Span<GradientPair> _out_gpair,
            common::Span<const bst_float> _preds,
            common::Span<const bst_float> _labels,
            common::Span<const bst_float> _weights) {
          const bst_float* preds_ptr = _preds.data();
          const bst_float* labels_ptr = _labels.data();
          const bst_float* weights_ptr = _weights.data();
          GradientPair* out_gpair_ptr = _out_gpair.data();
          const size_t begin = data_block_idx*block_size;
          const size_t end = std::min(ndata, begin + block_size);
          const float _scale_pos_weight = _additional_input[0];
          const bool _is_null_weight = _additional_input[1];

          for (size_t idx = begin; idx < end; ++idx) {
            bst_float p = Loss::PredTransform(preds_ptr[idx]);
            bst_float w = _is_null_weight ? 1.0f : weights_ptr[idx / n_targets];
            bst_float label = labels_ptr[idx];
            if (label == 1.0f) {
              w *= _scale_pos_weight;
            }
            out_gpair_ptr[idx] = GradientPair(Loss::FirstOrderGradient(p, label) * w,
                                              Loss::SecondOrderGradient(p, label) * w);
          }
        },
        common::Range{0, static_cast<int64_t>(n_data_blocks)}, nthreads, device)
        .Eval(&additional_input_, out_gpair->Data(), &preds, info.labels.Data(),
              &info.weights_);
  }

 public:
  [[nodiscard]] const char* DefaultEvalMetric() const override {
    return Loss::DefaultEvalMetric();
  }

  void PredTransform(HostDeviceVector<float> *io_preds) const override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<float> _preds) {
          _preds[_idx] = Loss::PredTransform(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())}, this->ctx_->Threads(),
        io_preds->Device())
        .Eval(io_preds);
  }

  void InitEstimation(MetaInfo const& info, linalg::Vector<float>* base_score) const override {
    if (std::abs(this->param_.scale_pos_weight - 1.0f) > kRtEps) {
      // Use newton method if `scale_pos_weight` is present. The alternative is to use
      // weighted mean, but we also need to take sample weight into account.
      FitIntercept::InitEstimation(info, base_score);
    } else {
      FitInterceptGlmLike::InitEstimation(info, base_score);
    }
  }

  void ProbToMargin(linalg::Vector<float>* base_score) const override {
    ProbToMarginImpl(
        this->ctx_, base_score, [] XGBOOST_DEVICE(float v) { return Loss::ProbToMargin(v); },
        [] XGBOOST_DEVICE(float v) { return Loss::CheckIntercept(v); }, Loss::InterceptErrorMsg);
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String(Loss::Name());
    out["reg_loss_param"] = ToJson(param_);
  }

  void LoadConfig(Json const& in) override {
    auto obj = get<Object const>(in);
    auto it = obj.find("reg_loss_param");
    if (it != obj.cend()) {
      FromJson(it->second, &param_);
    }
  }

 protected:
  RegLossParam param_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(RegLossParam);

XGBOOST_REGISTER_OBJECTIVE(SquaredLossRegression, LinearSquareLoss::Name())
.describe("Regression with squared error.")
.set_body([]() { return new RegLossObj<LinearSquareLoss>(); });

XGBOOST_REGISTER_OBJECTIVE(LogisticRegression, LogisticRegression::Name())
.describe("Logistic regression for probability regression task.")
.set_body([]() { return new RegLossObj<LogisticRegression>(); });

XGBOOST_REGISTER_OBJECTIVE(LogisticClassification, LogisticClassification::Name())
.describe("Logistic regression for binary classification task.")
.set_body([]() { return new RegLossObj<LogisticClassification>(); });

XGBOOST_REGISTER_OBJECTIVE(LogisticRaw, LogisticRaw::Name())
.describe("Logistic regression for classification, output score "
          "before logistic transformation.")
.set_body([]() { return new RegLossObj<LogisticRaw>(); });

XGBOOST_REGISTER_OBJECTIVE(GammaRegression, GammaDeviance::Name())
    .describe("Gamma regression using the gamma deviance loss with log link.")
    .set_body([]() { return new RegLossObj<GammaDeviance>(); });

// Deprecated functions
XGBOOST_REGISTER_OBJECTIVE(LinearRegression, "reg:linear")
.describe("Regression with squared error.")
.set_body([]() {
    LOG(WARNING) << "reg:linear is now deprecated in favor of reg:squarederror.";
    return new RegLossObj<LinearSquareLoss>(); });
// End deprecated

class SquaredLogErrorRegression : public FitIntercept {
 public:
  static auto Name() { return SquaredLogError::Name(); }

  void Configure(Args const&) override {}
  [[nodiscard]] ObjInfo Task() const override { return ObjInfo::kRegression; }
  [[nodiscard]] bst_target_t Targets(MetaInfo const& info) const override {
    return std::max(static_cast<std::size_t>(1), info.labels.Shape(1));
  }
  void GetGradient(HostDeviceVector<bst_float> const& preds, const MetaInfo& info,
                   std::int32_t iter, linalg::Matrix<GradientPair>* out_gpair) override {
    if (iter == 0) {
      ValidateLabel<SquaredLogError>(this->ctx_, info);
    }
    auto labels = info.labels.View(ctx_->Device());

    out_gpair->SetDevice(ctx_->Device());
    out_gpair->Reshape(info.num_row_, this->Targets(info));
    auto gpair = out_gpair->View(ctx_->Device());

    preds.SetDevice(ctx_->Device());
    auto predt = linalg::MakeTensorView(ctx_, &preds, info.num_row_, this->Targets(info));

    info.weights_.SetDevice(ctx_->Device());
    common::OptionalWeights weight{ctx_->IsCPU() ? info.weights_.ConstHostSpan()
                                                 : info.weights_.ConstDeviceSpan()};
    linalg::ElementWiseKernel(this->ctx_, labels,
                              [=] XGBOOST_DEVICE(std::size_t i, std::size_t j) mutable {
                                auto p = predt(i, j);
                                auto y = labels(i, j);
                                auto w = weight[i];
                                auto grad = SquaredLogError::FirstOrderGradient(p, y);
                                auto hess = SquaredLogError::SecondOrderGradient(p, y);
                                gpair(i) = {grad * w, hess * w};
                              });
  }
  [[nodiscard]] const char* DefaultEvalMetric() const override { return "rmsle"; }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String(Name());
  }
  void LoadConfig(Json const&) override {}
};

XGBOOST_REGISTER_OBJECTIVE(SquaredLogErrorRegression, SquaredLogErrorRegression::Name())
    .describe("Root mean squared log error.")
    .set_body([]() { return new SquaredLogErrorRegression(); });

class PseudoHuberRegression : public FitIntercept {
  PseudoHuberParam param_;

 public:
  void Configure(Args const& args) override { param_.UpdateAllowUnknown(args); }
  [[nodiscard]] ObjInfo Task() const override { return ObjInfo::kRegression; }
  [[nodiscard]] bst_target_t Targets(MetaInfo const& info) const override {
    return std::max(static_cast<std::size_t>(1), info.labels.Shape(1));
  }

  void GetGradient(HostDeviceVector<bst_float> const& preds, const MetaInfo& info, int /*iter*/,
                   linalg::Matrix<GradientPair>* out_gpair) override {
    CheckRegInputs(info, preds);
    auto slope = param_.huber_slope;
    CHECK_NE(slope, 0.0) << "slope for pseudo huber cannot be 0.";
    auto labels = info.labels.View(ctx_->Device());

    out_gpair->SetDevice(ctx_->Device());
    out_gpair->Reshape(info.num_row_, this->Targets(info));
    auto gpair = out_gpair->View(ctx_->Device());

    preds.SetDevice(ctx_->Device());
    auto predt = linalg::MakeTensorView(ctx_, &preds, info.num_row_, this->Targets(info));

    info.weights_.SetDevice(ctx_->Device());
    common::OptionalWeights weight{ctx_->IsCPU() ? info.weights_.ConstHostSpan()
                                                 : info.weights_.ConstDeviceSpan()};

    linalg::ElementWiseKernel(
        ctx_, labels, [=] XGBOOST_DEVICE(std::size_t i, std::size_t j) mutable {
          float z = predt(i, j) - labels(i, j);
          float scale_sqrt = std::sqrt(1 + common::Sqr(z) / common::Sqr(slope));
          float grad = z / scale_sqrt;

          auto scale = common::Sqr(slope) + common::Sqr(z);
          float hess = common::Sqr(slope) / (scale * scale_sqrt);

          auto w = weight[i];
          gpair(i) = {grad * w, hess * w};
        });
  }

  [[nodiscard]] const char* DefaultEvalMetric() const override { return "mphe"; }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("reg:pseudohubererror");
    out["pseudo_huber_param"] = ToJson(param_);
  }

  void LoadConfig(Json const& in) override {
    auto const& config = get<Object const>(in);
    if (config.find("pseudo_huber_param") == config.cend()) {
      // The parameter is added in 1.6.
      return;
    }
    FromJson(in["pseudo_huber_param"], &param_);
  }
  [[nodiscard]] Json DefaultMetricConfig() const override {
    CHECK(param_.GetInitialised());
    Json config{Object{}};
    config["name"] = String{this->DefaultEvalMetric()};
    config["pseudo_huber_param"] = ToJson(param_);
    return config;
  }
};

XGBOOST_REGISTER_OBJECTIVE(PseudoHuberRegression, "reg:pseudohubererror")
    .describe("Regression Pseudo Huber error.")
    .set_body([]() { return new PseudoHuberRegression(); });

// declare parameter
struct PoissonRegressionParam : public XGBoostParameter<PoissonRegressionParam> {
  float max_delta_step;
  DMLC_DECLARE_PARAMETER(PoissonRegressionParam) {
    DMLC_DECLARE_FIELD(max_delta_step).set_lower_bound(0.0f).set_default(0.7f)
        .describe("Maximum delta step we allow each weight estimation to be." \
                  " This parameter is required for possion regression.");
  }
};

// poisson regression for count
class PoissonRegression : public FitInterceptGlmLike {
 public:
  // declare functions
  void Configure(Args const& args) override { param_.UpdateAllowUnknown(args); }

  [[nodiscard]] ObjInfo Task() const override { return ObjInfo::kRegression; }

  void GetGradient(const HostDeviceVector<bst_float>& preds, const MetaInfo& info, int,
                   linalg::Matrix<GradientPair>* out_gpair) override {
    CHECK_NE(info.labels.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels.Size()) << "labels are not correctly provided";
    size_t const ndata = preds.Size();
    out_gpair->SetDevice(ctx_->Device());
    out_gpair->Reshape(info.num_row_, this->Targets(info));
    auto device = ctx_->Device();
    label_correct_.Resize(1);
    label_correct_.Fill(1);

    bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }
    bst_float max_delta_step = param_.max_delta_step;
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx];
          if (y < 0.0f) {
            _label_correct[0] = 0;
          }
          _out_gpair[_idx] = GradientPair{(expf(p) - y) * w,
                                          expf(p + max_delta_step) * w};
        },
        common::Range{0, static_cast<int64_t>(ndata)}, this->ctx_->Threads(), device).Eval(
            &label_correct_, out_gpair->Data(), &preds, info.labels.Data(), &info.weights_);
    // copy "label correct" flags back to host
    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag == 0) {
        LOG(FATAL) << "PoissonRegression: label must be nonnegative";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) const override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = expf(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())}, this->ctx_->Threads(),
        io_preds->Device())
        .Eval(io_preds);
  }
  void EvalTransform(HostDeviceVector<bst_float> *io_preds) override {
    PredTransform(io_preds);
  }
  void ProbToMargin(linalg::Vector<float>* base_score) const override {
    ProbToMarginImpl(this->ctx_, base_score, [] XGBOOST_DEVICE(float v) { return std::log(v); });
  }
  [[nodiscard]] const char* DefaultEvalMetric() const override {
    return "poisson-nloglik";
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("count:poisson");
    out["poisson_regression_param"] = ToJson(param_);
  }

  void LoadConfig(Json const& in) override {
    FromJson(in["poisson_regression_param"], &param_);
  }

 private:
  PoissonRegressionParam param_;
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(PoissonRegressionParam);

XGBOOST_REGISTER_OBJECTIVE(PoissonRegression, "count:poisson")
.describe("Poisson regression for count data.")
.set_body([]() { return new PoissonRegression(); });


// cox regression for survival data (negative values mean they are censored)
class CoxRegression : public FitIntercept {
 public:
  void Configure(Args const&) override {}
  [[nodiscard]] ObjInfo Task() const override { return ObjInfo::kRegression; }

  void GetGradient(const HostDeviceVector<bst_float>& preds, const MetaInfo& info, int,
                   linalg::Matrix<GradientPair>* out_gpair) override {
    CHECK_NE(info.labels.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels.Size()) << "labels are not correctly provided";
    const auto& preds_h = preds.HostVector();
    out_gpair->Reshape(info.num_row_, this->Targets(info));
    auto gpair = out_gpair->HostView();
    const std::vector<size_t> &label_order = info.LabelAbsSort(ctx_);

    const omp_ulong ndata = static_cast<omp_ulong>(preds_h.size()); // NOLINT(*)
    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }

    // pre-compute a sum
    double exp_p_sum = 0;  // we use double because we might need the precision with large datasets
    for (omp_ulong i = 0; i < ndata; ++i) {
      exp_p_sum += std::exp(preds_h[label_order[i]]);
    }

    // start calculating grad and hess
    const auto& labels = info.labels.HostView();
    double r_k = 0;
    double s_k = 0;
    double last_exp_p = 0.0;
    double last_abs_y = 0.0;
    double accumulated_sum = 0;
    for (omp_ulong i = 0; i < ndata; ++i) { // NOLINT(*)
      const size_t ind = label_order[i];
      const double p = preds_h[ind];
      const double exp_p = std::exp(p);
      const double w = info.GetWeight(ind);
      const double y = labels(ind);
      const double abs_y = std::abs(y);

      // only update the denominator after we move forward in time (labels are sorted)
      // this is Breslow's method for ties
      accumulated_sum += last_exp_p;
      if (last_abs_y < abs_y) {
        exp_p_sum -= accumulated_sum;
        accumulated_sum = 0;
      } else {
        CHECK(last_abs_y <= abs_y) << "CoxRegression: labels must be in sorted order, " <<
                                      "MetaInfo::LabelArgsort failed!";
      }

      if (y > 0) {
        r_k += 1.0/exp_p_sum;
        s_k += 1.0/(exp_p_sum*exp_p_sum);
      }

      const double grad = exp_p*r_k - static_cast<bst_float>(y > 0);
      const double hess = exp_p * r_k - exp_p * exp_p * s_k;
      gpair(ind) = GradientPair(grad * w, hess * w);

      last_abs_y = abs_y;
      last_exp_p = exp_p;
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) const override {
    std::vector<bst_float> &preds = io_preds->HostVector();
    const long ndata = static_cast<long>(preds.size()); // NOLINT(*)
    common::ParallelFor(ndata, ctx_->Threads(), [&](long j) { // NOLINT(*)
      preds[j] = std::exp(preds[j]);
    });
  }
  void EvalTransform(HostDeviceVector<bst_float> *io_preds) override {
    PredTransform(io_preds);
  }
  void ProbToMargin(linalg::Vector<float>* base_score) const override {
    ProbToMarginImpl(this->ctx_, base_score, [] XGBOOST_DEVICE(float v) { return std::log(v); });
  }
  [[nodiscard]] const char* DefaultEvalMetric() const override {
    return "cox-nloglik";
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("survival:cox");
  }
  void LoadConfig(Json const&) override {}
};

// register the objective function
XGBOOST_REGISTER_OBJECTIVE(CoxRegression, "survival:cox")
.describe("Cox regression for censored survival data (negative labels are considered censored).")
.set_body([]() { return new CoxRegression(); });


// declare parameter
struct TweedieRegressionParam : public XGBoostParameter<TweedieRegressionParam> {
  float tweedie_variance_power;
  DMLC_DECLARE_PARAMETER(TweedieRegressionParam) {
    DMLC_DECLARE_FIELD(tweedie_variance_power).set_range(1.0f, 2.0f).set_default(1.5f)
      .describe("Tweedie variance power.  Must be between in range [1, 2).");
  }
};

// tweedie regression
class TweedieRegression : public FitInterceptGlmLike {
 public:
  // declare functions
  void Configure(Args const& args) override {
    param_.UpdateAllowUnknown(args);
    std::ostringstream os;
    os << "tweedie-nloglik@" << param_.tweedie_variance_power;
    metric_ = os.str();
  }

  [[nodiscard]] ObjInfo Task() const override { return ObjInfo::kRegression; }

  void GetGradient(const HostDeviceVector<bst_float>& preds, const MetaInfo& info, std::int32_t,
                   linalg::Matrix<GradientPair>* out_gpair) override {
    CHECK_NE(info.labels.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels.Size()) << "labels are not correctly provided";
    const size_t ndata = preds.Size();
    out_gpair->SetDevice(ctx_->Device());
    out_gpair->Reshape(info.num_row_, this->Targets(info));

    auto device = ctx_->Device();
    label_correct_.Resize(1);
    label_correct_.Fill(1);

    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }

    const float rho = param_.tweedie_variance_power;
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx];
          if (y < 0.0f) {
            _label_correct[0] = 0;
          }
          bst_float grad = -y * expf((1 - rho) * p) + expf((2 - rho) * p);
          bst_float hess =
              -y * (1 - rho) * \
              std::exp((1 - rho) * p) + (2 - rho) * expf((2 - rho) * p);
          _out_gpair[_idx] = GradientPair(grad * w, hess * w);
        },
        common::Range{0, static_cast<int64_t>(ndata), 1}, this->ctx_->Threads(), device)
        .Eval(&label_correct_, out_gpair->Data(), &preds, info.labels.Data(), &info.weights_);

    // copy "label correct" flags back to host
    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag == 0) {
        LOG(FATAL) << "TweedieRegression: label must be nonnegative";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) const override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = expf(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())}, this->ctx_->Threads(),
        io_preds->Device())
        .Eval(io_preds);
  }
  void ProbToMargin(linalg::Vector<float>* base_score) const override {
    ProbToMarginImpl(this->ctx_, base_score, [] XGBOOST_DEVICE(float v) { return std::log(v); });
  }

  [[nodiscard]] const char* DefaultEvalMetric() const override {
    return metric_.c_str();
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("reg:tweedie");
    out["tweedie_regression_param"] = ToJson(param_);
  }
  void LoadConfig(Json const& in) override {
    FromJson(in["tweedie_regression_param"], &param_);
  }

 private:
  std::string metric_;
  TweedieRegressionParam param_;
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(TweedieRegressionParam);

XGBOOST_REGISTER_OBJECTIVE(TweedieRegression, "reg:tweedie")
.describe("Tweedie regression for insurance data.")
.set_body([]() { return new TweedieRegression(); });

class MeanAbsoluteError : public ObjFunction {
 public:
  void Configure(Args const&) override {}
  [[nodiscard]] ObjInfo Task() const override { return {ObjInfo::kRegression, true, true}; }
  [[nodiscard]] bst_target_t Targets(MetaInfo const& info) const override {
    return std::max(static_cast<std::size_t>(1), info.labels.Shape(1));
  }

  void GetGradient(HostDeviceVector<float> const& preds, const MetaInfo& info,
                   std::int32_t /*iter*/, linalg::Matrix<GradientPair>* out_gpair) override {
    CheckRegInputs(info, preds);
    auto labels = info.labels.View(ctx_->Device());

    out_gpair->SetDevice(ctx_->Device());
    out_gpair->Reshape(info.num_row_, this->Targets(info));
    auto gpair = out_gpair->View(ctx_->Device());

    preds.SetDevice(ctx_->Device());
    auto predt = linalg::MakeTensorView(ctx_, &preds, info.num_row_, this->Targets(info));
    info.weights_.SetDevice(ctx_->Device());
    common::OptionalWeights weight{ctx_->IsCPU() ? info.weights_.ConstHostSpan()
                                                 : info.weights_.ConstDeviceSpan()};

    linalg::ElementWiseKernel(
        ctx_, labels, [=] XGBOOST_DEVICE(std::size_t i, std::size_t j) mutable {
          auto sign = [](auto x) {
            return (x > static_cast<decltype(x)>(0)) - (x < static_cast<decltype(x)>(0));
          };
          auto y = labels(i, j);
          auto hess = weight[i];
          auto grad = sign(predt(i, j) - y) * hess;
          gpair(i, j) = GradientPair{grad, hess};
        });
  }

  void InitEstimation(MetaInfo const& info, linalg::Tensor<float, 1>* base_score) const override {
    CheckInitInputs(info);
    base_score->Reshape(this->Targets(info));

    double sum_weight{0.0};
    if (info.weights_.Empty()) {
      sum_weight = static_cast<double>(info.num_row_);
    } else {
      sum_weight = common::Reduce(ctx_, info.weights_);
    }

    if (info.num_row_ == 0) {
      auto out = base_score->HostView();
      std::fill(linalg::begin(out), linalg::end(out), 0.0f);
    } else {
      common::Median(ctx_, info.labels, info.weights_, base_score);
    }

    auto intercept = base_score->View(this->ctx_->Device());
    // weighted avg
    linalg::VecScaMul(this->ctx_, intercept, sum_weight);
    auto rc = collective::GlobalSum(ctx_, info, intercept, &sum_weight);
    collective::SafeColl(rc);

    if (common::CloseTo(sum_weight, 0.0)) {
      // Mostly for handling empty dataset test.
      LOG(WARNING) << "Sum of weights is close to 0.0, skipping base score estimation.";
      *base_score = linalg::Zeros<float>(ctx_, base_score->Shape(0));
      return;
    }

    linalg::VecScaDiv(this->ctx_, intercept, sum_weight);
  }

  void UpdateTreeLeaf(HostDeviceVector<bst_node_t> const& position, MetaInfo const& info,
                      float learning_rate, HostDeviceVector<float> const& prediction,
                      std::int32_t group_idx, RegTree* p_tree) const override {
    ::xgboost::obj::UpdateTreeLeaf(ctx_, position, group_idx, info, learning_rate, prediction, 0.5,
                                   p_tree);
  }

  [[nodiscard]] const char* DefaultEvalMetric() const override { return "mae"; }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("reg:absoluteerror");
  }

  void LoadConfig(Json const& in) override {
    CHECK_EQ(StringView{get<String const>(in["name"])}, StringView{"reg:absoluteerror"});
  }
};

XGBOOST_REGISTER_OBJECTIVE(MeanAbsoluteError, "reg:absoluteerror")
    .describe("Mean absoluate error.")
    .set_body([]() { return new MeanAbsoluteError(); });
}  // namespace xgboost::obj
