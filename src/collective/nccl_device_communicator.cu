
#include <hip/hip_runtime.h>
/*!
 * Copyright 2023 XGBoost contributors
 */
#if defined(XGBOOST_USE_NCCL)
#include "nccl_device_communicator.cuh"

namespace xgboost {
namespace collective {

NcclDeviceCommunicator::NcclDeviceCommunicator(int device_ordinal, bool needs_sync)
    : device_ordinal_{device_ordinal},
      needs_sync_{needs_sync},
      world_size_{GetWorldSize()},
      rank_{GetRank()} {
  if (device_ordinal_ < 0) {
    LOG(FATAL) << "Invalid device ordinal: " << device_ordinal_;
  }
  if (world_size_ == 1) {
    return;
  }

  std::vector<uint64_t> uuids(world_size_ * kUuidLength, 0);
  auto s_uuid = xgboost::common::Span<uint64_t>{uuids.data(), uuids.size()};
  auto s_this_uuid = s_uuid.subspan(rank_ * kUuidLength, kUuidLength);
  GetCudaUUID(s_this_uuid);

  // TODO(rongou): replace this with allgather.
  Allreduce(uuids.data(), uuids.size(), DataType::kUInt64, Operation::kSum);

  std::vector<xgboost::common::Span<uint64_t, kUuidLength>> converted(world_size_);
  size_t j = 0;
  for (size_t i = 0; i < uuids.size(); i += kUuidLength) {
    converted[j] = xgboost::common::Span<uint64_t, kUuidLength>{uuids.data() + i, kUuidLength};
    j++;
  }

  auto iter = std::unique(converted.begin(), converted.end());
  auto n_uniques = std::distance(converted.begin(), iter);

  CHECK_EQ(n_uniques, world_size_)
      << "Multiple processes within communication group running on same CUDA "
      << "device is not supported. " << PrintUUID(s_this_uuid) << "\n";

  nccl_unique_id_ = GetUniqueId();
  dh::safe_cuda(cudaSetDevice(device_ordinal_));
  dh::safe_nccl(ncclCommInitRank(&nccl_comm_, world_size_, nccl_unique_id_, rank_));
}

NcclDeviceCommunicator::~NcclDeviceCommunicator() {
  if (world_size_ == 1) {
    return;
  }
  if (nccl_comm_) {
    dh::safe_nccl(ncclCommDestroy(nccl_comm_));
  }
  if (xgboost::ConsoleLogger::ShouldLog(xgboost::ConsoleLogger::LV::kDebug)) {
    LOG(CONSOLE) << "======== NCCL Statistics========";
    LOG(CONSOLE) << "AllReduce calls: " << allreduce_calls_;
    LOG(CONSOLE) << "AllReduce total MiB communicated: " << allreduce_bytes_ / 1048576;
  }
}

namespace {
ncclDataType_t GetNcclDataType(DataType const &data_type) {
  ncclDataType_t result{ncclInt8};
  switch (data_type) {
    case DataType::kInt8:
      result = ncclInt8;
      break;
    case DataType::kUInt8:
      result = ncclUint8;
      break;
    case DataType::kInt32:
      result = ncclInt32;
      break;
    case DataType::kUInt32:
      result = ncclUint32;
      break;
    case DataType::kInt64:
      result = ncclInt64;
      break;
    case DataType::kUInt64:
      result = ncclUint64;
      break;
    case DataType::kFloat:
      result = ncclFloat;
      break;
    case DataType::kDouble:
      result = ncclDouble;
      break;
    default:
      LOG(FATAL) << "Unknown data type.";
  }
  return result;
}

bool IsBitwiseOp(Operation const &op) {
  return op == Operation::kBitwiseAND || op == Operation::kBitwiseOR ||
         op == Operation::kBitwiseXOR;
}

ncclRedOp_t GetNcclRedOp(Operation const &op) {
  ncclRedOp_t result{ncclMax};
  switch (op) {
    case Operation::kMax:
      result = ncclMax;
      break;
    case Operation::kMin:
      result = ncclMin;
      break;
    case Operation::kSum:
      result = ncclSum;
      break;
    default:
      LOG(FATAL) << "Unsupported reduce operation.";
  }
  return result;
}

template <typename Func>
void RunBitwiseAllreduce(char *out_buffer, char const *device_buffer, Func func, int world_size,
                         std::size_t size) {
  dh::LaunchN(size, [=] __device__(std::size_t idx) {
    auto result = device_buffer[idx];
    for (auto rank = 1; rank < world_size; rank++) {
      result = func(result, device_buffer[rank * size + idx]);
    }
    out_buffer[idx] = result;
  });
}
}  // anonymous namespace

void NcclDeviceCommunicator::BitwiseAllReduce(void *send_receive_buffer, std::size_t count,
                                              DataType data_type, Operation op) {
  auto const size = count * GetTypeSize(data_type);
  dh::caching_device_vector<char> buffer(size * world_size_);
  auto *device_buffer = buffer.data().get();

  // First gather data from all the workers.
  dh::safe_nccl(ncclAllGather(send_receive_buffer, device_buffer, count, GetNcclDataType(data_type),
                              nccl_comm_, dh::DefaultStream()));
  if (needs_sync_) {
    dh::DefaultStream().Sync();
  }

  // Then reduce locally.
  auto *out_buffer = static_cast<char *>(send_receive_buffer);
  switch (op) {
    case Operation::kBitwiseAND:
      RunBitwiseAllreduce(out_buffer, device_buffer, thrust::bit_and<char>(), world_size_, size);
      break;
    case Operation::kBitwiseOR:
      RunBitwiseAllreduce(out_buffer, device_buffer, thrust::bit_or<char>(), world_size_, size);
      break;
    case Operation::kBitwiseXOR:
      RunBitwiseAllreduce(out_buffer, device_buffer, thrust::bit_xor<char>(), world_size_, size);
      break;
    default:
      LOG(FATAL) << "Not a bitwise reduce operation.";
  }
}

void NcclDeviceCommunicator::AllReduce(void *send_receive_buffer, std::size_t count,
                                       DataType data_type, Operation op) {
  if (world_size_ == 1) {
    return;
  }

  dh::safe_cuda(cudaSetDevice(device_ordinal_));
  if (IsBitwiseOp(op)) {
    BitwiseAllReduce(send_receive_buffer, count, data_type, op);
  } else {
    dh::safe_nccl(ncclAllReduce(send_receive_buffer, send_receive_buffer, count,
                                GetNcclDataType(data_type), GetNcclRedOp(op), nccl_comm_,
                                dh::DefaultStream()));
  }
  allreduce_bytes_ += count * GetTypeSize(data_type);
  allreduce_calls_ += 1;
}

void NcclDeviceCommunicator::AllGatherV(void const *send_buffer, size_t length_bytes,
                                        std::vector<std::size_t> *segments,
                                        dh::caching_device_vector<char> *receive_buffer) {
  if (world_size_ == 1) {
    return;
  }

  dh::safe_cuda(cudaSetDevice(device_ordinal_));

  segments->clear();
  segments->resize(world_size_, 0);
  segments->at(rank_) = length_bytes;
  Allreduce(segments->data(), segments->size(), DataType::kUInt64, Operation::kMax);
  auto total_bytes = std::accumulate(segments->cbegin(), segments->cend(), 0UL);
  receive_buffer->resize(total_bytes);

  size_t offset = 0;
  dh::safe_nccl(ncclGroupStart());
  for (int32_t i = 0; i < world_size_; ++i) {
    size_t as_bytes = segments->at(i);
    dh::safe_nccl(ncclBroadcast(send_buffer, receive_buffer->data().get() + offset, as_bytes,
                                ncclChar, i, nccl_comm_, dh::DefaultStream()));
    offset += as_bytes;
  }
  dh::safe_nccl(ncclGroupEnd());
}

void NcclDeviceCommunicator::Synchronize() {
  if (world_size_ == 1) {
    return;
  }
  dh::safe_cuda(cudaSetDevice(device_ordinal_));
  dh::DefaultStream().Sync();
}

}  // namespace collective
}  // namespace xgboost
#endif
