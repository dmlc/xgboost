
#include <hip/hip_runtime.h>
/*!
 * Copyright 2023 XGBoost contributors
 */
#if defined(XGBOOST_USE_NCCL)
#include "nccl_device_communicator.cuh"

namespace xgboost {
namespace collective {

NcclDeviceCommunicator::NcclDeviceCommunicator(int device_ordinal, Communicator *communicator)
    : device_ordinal_{device_ordinal}, communicator_{communicator} {
  if (device_ordinal_ < 0) {
    LOG(FATAL) << "Invalid device ordinal: " << device_ordinal_;
  }
  if (communicator_ == nullptr) {
    LOG(FATAL) << "Communicator cannot be null.";
  }

  int32_t const rank = communicator_->GetRank();
  int32_t const world = communicator_->GetWorldSize();

  if (world == 1) {
    return;
  }

  std::vector<uint64_t> uuids(world * kUuidLength, 0);
  auto s_uuid = xgboost::common::Span<uint64_t>{uuids.data(), uuids.size()};
  auto s_this_uuid = s_uuid.subspan(rank * kUuidLength, kUuidLength);
  GetCudaUUID(s_this_uuid);

  // TODO(rongou): replace this with allgather.
  communicator_->AllReduce(uuids.data(), uuids.size(), DataType::kUInt64, Operation::kSum);

  std::vector<xgboost::common::Span<uint64_t, kUuidLength>> converted(world);
  size_t j = 0;
  for (size_t i = 0; i < uuids.size(); i += kUuidLength) {
    converted[j] = xgboost::common::Span<uint64_t, kUuidLength>{uuids.data() + i, kUuidLength};
    j++;
  }

  auto iter = std::unique(converted.begin(), converted.end());
  auto n_uniques = std::distance(converted.begin(), iter);

  CHECK_EQ(n_uniques, world)
      << "Multiple processes within communication group running on same CUDA "
      << "device is not supported. " << PrintUUID(s_this_uuid) << "\n";

  nccl_unique_id_ = GetUniqueId();
  dh::safe_cuda(cudaSetDevice(device_ordinal_));
  dh::safe_nccl(ncclCommInitRank(&nccl_comm_, world, nccl_unique_id_, rank));
  dh::safe_cuda(cudaStreamCreate(&cuda_stream_));
}

NcclDeviceCommunicator::~NcclDeviceCommunicator() {
  if (communicator_->GetWorldSize() == 1) {
    return;
  }
  if (cuda_stream_) {
    dh::safe_cuda(cudaStreamDestroy(cuda_stream_));
  }
  if (nccl_comm_) {
    dh::safe_nccl(ncclCommDestroy(nccl_comm_));
  }
  if (xgboost::ConsoleLogger::ShouldLog(xgboost::ConsoleLogger::LV::kDebug)) {
    LOG(CONSOLE) << "======== NCCL Statistics========";
    LOG(CONSOLE) << "AllReduce calls: " << allreduce_calls_;
    LOG(CONSOLE) << "AllReduce total MiB communicated: " << allreduce_bytes_ / 1048576;
  }
}

namespace {
ncclDataType_t GetNcclDataType(DataType const &data_type) {
  ncclDataType_t result;
  switch (data_type) {
    case DataType::kInt8:
      result = ncclInt8;
      break;
    case DataType::kUInt8:
      result = ncclUint8;
      break;
    case DataType::kInt32:
      result = ncclInt32;
      break;
    case DataType::kUInt32:
      result = ncclUint32;
      break;
    case DataType::kInt64:
      result = ncclInt64;
      break;
    case DataType::kUInt64:
      result = ncclUint64;
      break;
    case DataType::kFloat:
      result = ncclFloat;
      break;
    case DataType::kDouble:
      result = ncclDouble;
      break;
    default:
      LOG(FATAL) << "Unknown data type.";
  }
  return result;
}

bool IsBitwiseOp(Operation const &op) {
  return op == Operation::kBitwiseAND || op == Operation::kBitwiseOR ||
         op == Operation::kBitwiseXOR;
}

ncclRedOp_t GetNcclRedOp(Operation const &op) {
  ncclRedOp_t result;
  switch (op) {
    case Operation::kMax:
      result = ncclMax;
      break;
    case Operation::kMin:
      result = ncclMin;
      break;
    case Operation::kSum:
      result = ncclSum;
      break;
    default:
      LOG(FATAL) << "Unsupported reduce operation.";
  }
  return result;
}

template <typename Func>
void RunBitwiseAllreduce(char *out_buffer, char const *device_buffer, Func func, int world_size,
                         std::size_t size, cudaStream_t stream) {
  dh::LaunchN(size, stream, [=] __device__(std::size_t idx) {
    out_buffer[idx] = device_buffer[idx];
    for (auto rank = 1; rank < world_size; rank++) {
      out_buffer[idx] = func(out_buffer[idx], device_buffer[rank * size + idx]);
    }
  });
}
}  // anonymous namespace

void NcclDeviceCommunicator::BitwiseAllReduce(void *send_receive_buffer, std::size_t count,
                                              DataType data_type, Operation op) {
  auto const world_size = communicator_->GetWorldSize();
  auto const size = count * GetTypeSize(data_type);
  dh::caching_device_vector<char> buffer(size * world_size);
  auto *device_buffer = buffer.data().get();

  // First gather data from all the workers.
  dh::safe_nccl(ncclAllGather(send_receive_buffer, device_buffer, count, GetNcclDataType(data_type),
                              nccl_comm_, cuda_stream_));

  // Then reduce locally.
  auto *out_buffer = static_cast<char *>(send_receive_buffer);
  switch (op) {
    case Operation::kBitwiseAND:
      RunBitwiseAllreduce(out_buffer, device_buffer, thrust::bit_and<char>(), world_size, size,
                          cuda_stream_);
      break;
    case Operation::kBitwiseOR:
      RunBitwiseAllreduce(out_buffer, device_buffer, thrust::bit_or<char>(), world_size, size,
                          cuda_stream_);
      break;
    case Operation::kBitwiseXOR:
      RunBitwiseAllreduce(out_buffer, device_buffer, thrust::bit_xor<char>(), world_size, size,
                          cuda_stream_);
      break;
    default:
      LOG(FATAL) << "Not a bitwise reduce operation.";
  }
}

void NcclDeviceCommunicator::AllReduce(void *send_receive_buffer, std::size_t count,
                                       DataType data_type, Operation op) {
  if (communicator_->GetWorldSize() == 1) {
    return;
  }

  dh::safe_cuda(cudaSetDevice(device_ordinal_));
  if (IsBitwiseOp(op)) {
    BitwiseAllReduce(send_receive_buffer, count, data_type, op);
  } else {
    dh::safe_nccl(ncclAllReduce(send_receive_buffer, send_receive_buffer, count,
                                GetNcclDataType(data_type), GetNcclRedOp(op), nccl_comm_,
                                cuda_stream_));
  }
  allreduce_bytes_ += count * GetTypeSize(data_type);
  allreduce_calls_ += 1;
}

void NcclDeviceCommunicator::AllGatherV(void const *send_buffer, size_t length_bytes,
                                        std::vector<std::size_t> *segments,
                                        dh::caching_device_vector<char> *receive_buffer) {
  if (communicator_->GetWorldSize() == 1) {
    return;
  }

  dh::safe_cuda(cudaSetDevice(device_ordinal_));
  int const world_size = communicator_->GetWorldSize();
  int const rank = communicator_->GetRank();

  segments->clear();
  segments->resize(world_size, 0);
  segments->at(rank) = length_bytes;
  communicator_->AllReduce(segments->data(), segments->size(), DataType::kUInt64, Operation::kMax);
  auto total_bytes = std::accumulate(segments->cbegin(), segments->cend(), 0UL);
  receive_buffer->resize(total_bytes);

  size_t offset = 0;
  dh::safe_nccl(ncclGroupStart());
  for (int32_t i = 0; i < world_size; ++i) {
    size_t as_bytes = segments->at(i);
    dh::safe_nccl(ncclBroadcast(send_buffer, receive_buffer->data().get() + offset, as_bytes,
                                ncclChar, i, nccl_comm_, cuda_stream_));
    offset += as_bytes;
  }
  dh::safe_nccl(ncclGroupEnd());
}

void NcclDeviceCommunicator::Synchronize() {
  if (communicator_->GetWorldSize() == 1) {
    return;
  }
  dh::safe_cuda(cudaSetDevice(device_ordinal_));
  dh::safe_cuda(cudaStreamSynchronize(cuda_stream_));
}

}  // namespace collective
}  // namespace xgboost
#endif
