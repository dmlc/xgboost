/*!
 * Copyright 2021 XGBoost contributors
 */
#include <gtest/gtest.h>
#include <xgboost/data.h>

#include "../../../src/data/ellpack_page.cuh"
#include "../../../src/data/sparse_page_source.h"
#include "../filesystem.h"  // dmlc::TemporaryDirectory
#include "../helpers.h"

namespace xgboost {
namespace data {
TEST(EllpackPageRawFormat, IO) {
  std::unique_ptr<SparsePageFormat<EllpackPage>> format{CreatePageFormat<EllpackPage>("raw")};

  auto m = RandomDataGenerator{100, 14, 0.5}.GenerateDMatrix();
  dmlc::TemporaryDirectory tmpdir;
  std::string path = tmpdir.path + "/ellpack.page";

  {
    std::unique_ptr<dmlc::Stream> fo{dmlc::Stream::Create(path.c_str(), "w")};
    for (auto const &ellpack : m->GetBatches<EllpackPage>({0, 256})) {
      format->Write(ellpack, fo.get());
    }
  }

  EllpackPage page;
  std::unique_ptr<dmlc::SeekStream> fi{dmlc::SeekStream::CreateForRead(path.c_str())};
  format->Read(&page, fi.get());

  for (auto const &ellpack : m->GetBatches<EllpackPage>({0, 256})) {
    auto loaded = page.Impl();
    auto orig = ellpack.Impl();
    ASSERT_EQ(loaded->Cuts().Ptrs(), orig->Cuts().Ptrs());
    ASSERT_EQ(loaded->Cuts().MinValues(), orig->Cuts().MinValues());
    ASSERT_EQ(loaded->Cuts().Values(), orig->Cuts().Values());
    ASSERT_EQ(loaded->base_rowid, orig->base_rowid);
    ASSERT_EQ(loaded->row_stride, orig->row_stride);
    ASSERT_EQ(loaded->gidx_buffer.HostVector(), orig->gidx_buffer.HostVector());
  }
}
}  // namespace data
}  // namespace xgboost
