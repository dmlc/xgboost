#include "hip/hip_runtime.h"
// Copyright by Contributors
#include <dmlc/filesystem.h>
#include <xgboost/data.h>
#include "../../../src/data/simple_dmatrix.h"

#include <thrust/sequence.h>
#include "../../../src/data/device_adapter.cuh"
#include "../helpers.h"
#include "test_array_interface.h"
#include "../../../src/data/array_interface.h"

using namespace xgboost;  // NOLINT

TEST(SimpleDMatrix, FromColumnarDenseBasic) {
  constexpr size_t kRows{16};
  std::vector<Json> columns;
  thrust::device_vector<double> d_data_0(kRows);
  thrust::device_vector<uint32_t> d_data_1(kRows);

  columns.emplace_back(GenerateDenseColumn<double>("<f8", kRows, &d_data_0));
  columns.emplace_back(GenerateDenseColumn<uint32_t>("<u4", kRows, &d_data_1));

  Json column_arr{columns};

  std::stringstream ss;
  Json::Dump(column_arr, &ss);
  std::string str = ss.str();

  data::CudfAdapter adapter(str);
  data::SimpleDMatrix dmat(&adapter, std::numeric_limits<float>::quiet_NaN(),
                           -1);
  EXPECT_EQ(dmat.Info().num_col_, 2);
  EXPECT_EQ(dmat.Info().num_row_, 16);
  EXPECT_EQ(dmat.Info().num_nonzero_, 32);
}

void TestDenseColumn(DMatrix* dmat, size_t n_rows, size_t n_cols) {
  for (auto& batch : dmat->GetBatches<SparsePage>()) {
    for (auto i = 0ull; i < batch.Size(); i++) {
      auto inst = batch[i];
      for (auto j = 0ull; j < inst.size(); j++) {
        EXPECT_EQ(inst[j].fvalue, i * 2);
        EXPECT_EQ(inst[j].index, j);
      }
    }
  }
  ASSERT_EQ(dmat->Info().num_row_, n_rows);
  ASSERT_EQ(dmat->Info().num_col_, n_cols);
}

TEST(SimpleDMatrix, FromColumnarDense) {
  constexpr size_t kRows{16};
  constexpr size_t kCols{2};
  std::vector<Json> columns;
  thrust::device_vector<float> d_data_0(kRows);
  thrust::device_vector<int32_t> d_data_1(kRows);
  columns.emplace_back(GenerateDenseColumn<float>("<f4", kRows, &d_data_0));
  columns.emplace_back(GenerateDenseColumn<int32_t>("<i4", kRows, &d_data_1));

  Json column_arr{columns};

  std::stringstream ss;
  Json::Dump(column_arr, &ss);
  std::string str = ss.str();

  // no missing value
  {
    data::CudfAdapter adapter(str);
    data::SimpleDMatrix dmat(&adapter, std::numeric_limits<float>::quiet_NaN(),
                             -1);
    TestDenseColumn(&dmat, kRows, kCols);
  }

  // with missing value specified
  {
    data::CudfAdapter adapter(str);
    data::SimpleDMatrix dmat(&adapter, 4.0, -1);

    ASSERT_EQ(dmat.Info().num_row_, kRows);
    ASSERT_EQ(dmat.Info().num_col_, kCols);
    ASSERT_EQ(dmat.Info().num_nonzero_, kCols * kRows - 2);
  }

  {
    // no missing value, but has NaN
    d_data_0[3] = std::numeric_limits<float>::quiet_NaN();
    ASSERT_TRUE(std::isnan(d_data_0[3]));  // removes 6.0
    data::CudfAdapter adapter(str);
    data::SimpleDMatrix dmat(&adapter, std::numeric_limits<float>::quiet_NaN(),
                             -1);
    ASSERT_EQ(dmat.Info().num_nonzero_, kRows * kCols - 1);
    ASSERT_EQ(dmat.Info().num_row_, kRows);
    ASSERT_EQ(dmat.Info().num_col_, kCols);
  }
}

TEST(SimpleDMatrix, FromColumnarWithEmptyRows) {
  constexpr size_t kRows = 102;
  constexpr size_t kCols = 24;

  std::vector<Json> v_columns(kCols);
  std::vector<dh::device_vector<float>> columns_data(kCols);
  std::vector<dh::device_vector<RBitField8::value_type>> column_bitfields(
      kCols);

  RBitField8::value_type constexpr kUCOne = 1;

  for (size_t i = 0; i < kCols; ++i) {
    auto& col = v_columns[i];
    col = Object();
    auto& data = columns_data[i];
    data.resize(kRows);
    thrust::sequence(data.begin(), data.end(), 0);
    dh::safe_cuda(hipDeviceSynchronize());
    dh::safe_cuda(hipGetLastError());

    ASSERT_EQ(data.size(), kRows);

    auto p_d_data = raw_pointer_cast(data.data());
    std::vector<Json> j_data{
        Json(Integer(reinterpret_cast<Integer::Int>(p_d_data))),
        Json(Boolean(false))};
    col["data"] = j_data;
    std::vector<Json> j_shape{Json(Integer(static_cast<Integer::Int>(kRows)))};
    col["shape"] = Array(j_shape);
    col["version"] = Integer(static_cast<Integer::Int>(1));
    col["typestr"] = String("<f4");

    // Construct the mask object.
    col["mask"] = Object();
    auto& j_mask = col["mask"];
    j_mask["version"] = Integer(static_cast<Integer::Int>(1));
    auto& mask_storage = column_bitfields[i];
    mask_storage.resize(16);  // 16 bytes

    mask_storage[0] = ~(kUCOne << 2);  // 3^th row is missing
    mask_storage[1] = ~(kUCOne << 3);  // 12^th row is missing
    size_t last_ind = 12;
    mask_storage[last_ind] = ~(kUCOne << 5);
    std::set<size_t> missing_row_index{0, 1, last_ind};

    for (size_t j = 0; j < mask_storage.size(); ++j) {
      if (missing_row_index.find(j) == missing_row_index.cend()) {
        // all other rows are valid
        mask_storage[j] = ~0;
      }
    }

    j_mask["data"] = std::vector<Json>{
        Json(
            Integer(reinterpret_cast<Integer::Int>(mask_storage.data().get()))),
        Json(Boolean(false))};
    j_mask["shape"] = Array(
        std::vector<Json>{Json(Integer(static_cast<Integer::Int>(kRows)))});
    j_mask["typestr"] = String("|i1");
  }

  Json column_arr{Array(v_columns)};
  std::stringstream ss;
  Json::Dump(column_arr, &ss);
  std::string str = ss.str();
  data::CudfAdapter adapter(str);
  data::SimpleDMatrix dmat(&adapter, std::numeric_limits<float>::quiet_NaN(),
                           -1);

  for (auto& batch : dmat.GetBatches<SparsePage>()) {
    for (auto i = 0ull; i < batch.Size(); i++) {
      auto inst = batch[i];
      for (auto j = 0ull; j < inst.size(); j++) {
        EXPECT_EQ(inst[j].fvalue, i);
        EXPECT_EQ(inst[j].index, j);
      }
    }
  }
  ASSERT_EQ(dmat.Info().num_nonzero_, (kRows - 3) * kCols);
  ASSERT_EQ(dmat.Info().num_row_, kRows);
  ASSERT_EQ(dmat.Info().num_col_, kCols);
}

TEST(SimpleCSRSource, FromColumnarSparse) {
  constexpr size_t kRows = 32;
  constexpr size_t kCols = 2;
  RBitField8::value_type constexpr kUCOne = 1;

  std::vector<dh::device_vector<float>> columns_data(kCols);
  std::vector<dh::device_vector<RBitField8::value_type>> column_bitfields(kCols);

  {
    // column 0
    auto& mask = column_bitfields[0];
    mask.resize(8);

    for (size_t j = 0; j < mask.size(); ++j) {
      mask[j] = ~0;
    }
    // the 2^th entry of first column is invalid
    // [0 0 0 0 0 1 0 0]
    mask[0] = ~(kUCOne << 2);
  }
  {
    // column 1
    auto& mask = column_bitfields[1];
    mask.resize(8);

    for (size_t j = 0; j < mask.size(); ++j) {
      mask[j] = ~0;
    }
    // the 19^th entry of second column is invalid
    // [~0~], [~0~], [0 0 0 0 1 0 0 0]
    mask[2] = ~(kUCOne << 3);
  }

  for (size_t c = 0; c < kCols; ++c) {
    columns_data[c].resize(kRows);
    thrust::sequence(columns_data[c].begin(), columns_data[c].end(), 0);
  }

  std::vector<Json> j_columns(kCols);

  for (size_t c = 0; c < kCols; ++c) {
    auto& column = j_columns[c];
    column = Object();
    column["version"] = Integer(static_cast<Integer::Int>(1));
    column["typestr"] = String("<f4");
    auto p_d_data = raw_pointer_cast(columns_data[c].data());
    std::vector<Json> j_data {
      Json(Integer(reinterpret_cast<Integer::Int>(p_d_data))),
          Json(Boolean(false))};
    column["data"] = j_data;
    std::vector<Json> j_shape {Json(Integer(static_cast<Integer::Int>(kRows)))};
    column["shape"] = Array(j_shape);
    column["version"] = Integer(static_cast<Integer::Int>(1));
    column["typestr"] = String("<f4");

    column["mask"] = Object();
    auto& j_mask = column["mask"];
    j_mask["version"] = Integer(static_cast<Integer::Int>(1));
    j_mask["data"] = std::vector<Json>{
      Json(Integer(reinterpret_cast<Integer::Int>(column_bitfields[c].data().get()))),
      Json(Boolean(false))};
    j_mask["shape"] = Array(std::vector<Json>{Json(Integer(static_cast<Integer::Int>(kRows)))});
    j_mask["typestr"] = String("|i1");
  }

  Json column_arr {Array(j_columns)};

  std::stringstream ss;
  Json::Dump(column_arr, &ss);
  std::string str = ss.str();

  {
    data::CudfAdapter adapter(str);
    data::SimpleDMatrix dmat(&adapter, std::numeric_limits<float>::quiet_NaN(), -1);

    ASSERT_EQ(dmat.Info().num_row_, kRows);
    ASSERT_EQ(dmat.Info().num_nonzero_, (kRows*kCols)-2);
  }

  {
    data::CudfAdapter adapter(str);
    data::SimpleDMatrix dmat(&adapter, 2.0, -1);
    for (auto& batch : dmat.GetBatches<SparsePage>()) {
      for (auto i = 0ull; i < batch.Size(); i++) {
        auto inst = batch[i];
        for (auto e : inst) {
          ASSERT_NE(e.fvalue, 2.0);
        }
      }
    }
  }

  {
    // no missing value, but has NaN
    data::CudfAdapter adapter(str);
    columns_data[0][4] = std::numeric_limits<float>::quiet_NaN();  // 0^th column 4^th row
    data::SimpleDMatrix dmat(&adapter, std::numeric_limits<float>::quiet_NaN(),
                             -1);
    ASSERT_TRUE(std::isnan(columns_data[0][4]));

    // Two invalid entries and one NaN, in CSC
    // 0^th column: 0, 1, 4, 5, 6, ..., kRows
    // 1^th column: 0, 1, 2, 3, ..., 19, 21, ..., kRows
    ASSERT_EQ(dmat.Info().num_nonzero_, kRows * kCols - 3);
  }
}


TEST(SimpleDMatrix, FromColumnarSparseBasic) {
  constexpr size_t kRows{16};
  std::vector<Json> columns;
  thrust::device_vector<double> d_data_0(kRows);
  thrust::device_vector<uint32_t> d_data_1(kRows);

  columns.emplace_back(GenerateSparseColumn<double>("<f8", kRows, &d_data_0));
  columns.emplace_back(GenerateSparseColumn<uint32_t>("<u4", kRows, &d_data_1));

  Json column_arr{columns};

  std::stringstream ss;
  Json::Dump(column_arr, &ss);
  std::string str = ss.str();

  data::CudfAdapter adapter(str);
  data::SimpleDMatrix dmat(&adapter, std::numeric_limits<float>::quiet_NaN(),
                           -1);
  EXPECT_EQ(dmat.Info().num_col_, 2);
  EXPECT_EQ(dmat.Info().num_row_, 16);
  EXPECT_EQ(dmat.Info().num_nonzero_, 32);

  for (auto& batch : dmat.GetBatches<SparsePage>()) {
    for (auto i = 0ull; i < batch.Size(); i++) {
      auto inst = batch[i];
      for (auto j = 0ull; j < inst.size(); j++) {
        EXPECT_EQ(inst[j].fvalue, i * 2);
        EXPECT_EQ(inst[j].index, j);
      }
    }
  }
}


TEST(SimpleDMatrix, FromCupy){
  int rows = 50;
  int cols = 10;
  thrust::device_vector< float> data(rows*cols);
  auto json_array_interface = Generate2dArrayInterface(rows, cols, "<f4", &data);
  std::stringstream ss;
  Json::Dump(json_array_interface, &ss);
  std::string str = ss.str();
  data::CupyAdapter adapter(str);
  data::SimpleDMatrix dmat(&adapter, -1, 1);
  EXPECT_EQ(dmat.Info().num_col_, cols);
  EXPECT_EQ(dmat.Info().num_row_, rows);
  EXPECT_EQ(dmat.Info().num_nonzero_, rows*cols);

  for (auto& batch : dmat.GetBatches<SparsePage>()) {
    for (auto i = 0ull; i < batch.Size(); i++) {
      auto inst = batch[i];
      for (auto j = 0ull; j < inst.size(); j++) {
        EXPECT_EQ(inst[j].fvalue, i * cols + j);
        EXPECT_EQ(inst[j].index, j);
      }
    }
  }
}

TEST(SimpleDMatrix, FromCupySparse){
  int rows = 2;
  int cols = 2;
  thrust::device_vector< float> data(rows*cols);
  auto json_array_interface = Generate2dArrayInterface(rows, cols, "<f4", &data);
  data[1] = std::numeric_limits<float>::quiet_NaN();
  data[2] = std::numeric_limits<float>::quiet_NaN();
  std::stringstream ss;
  Json::Dump(json_array_interface, &ss);
  std::string str = ss.str();
  data::CupyAdapter adapter(str);
  data::SimpleDMatrix dmat(&adapter, -1, 1);
  EXPECT_EQ(dmat.Info().num_col_, cols);
  EXPECT_EQ(dmat.Info().num_row_, rows);
  EXPECT_EQ(dmat.Info().num_nonzero_, rows * cols - 2);
  auto& batch = *dmat.GetBatches<SparsePage>().begin();
  auto inst0 = batch[0];
  auto inst1 = batch[1];
  EXPECT_EQ(batch[0].size(), 1);
  EXPECT_EQ(batch[1].size(), 1);
  EXPECT_EQ(batch[0][0].fvalue, 0.0f);
  EXPECT_EQ(batch[0][0].index, 0);
  EXPECT_EQ(batch[1][0].fvalue, 3.0f);
  EXPECT_EQ(batch[1][0].index, 1);
}
