/*! Copyright 2019-2021 by XGBoost Contributors */

#include <gtest/gtest.h>
#include <xgboost/data.h>
#include <xgboost/json.h>
#include <xgboost/generic_parameters.h>
#include <thrust/device_vector.h>
#include "test_array_interface.h"
#include "../../../src/common/device_helpers.cuh"

#include "test_metainfo.h"

namespace xgboost {

template <typename T>
std::string PrepareData(std::string typestr, thrust::device_vector<T>* out, const size_t kRows=16) {
  out->resize(kRows);
  auto& d_data = *out;

  for (size_t i = 0; i < d_data.size(); ++i) {
    d_data[i] = i * 2.0;
  }

  Json column { Object() };

  std::vector<Json> j_shape {Json(Integer(static_cast<Integer::Int>(kRows)))};
  column["shape"] = Array(j_shape);
  column["strides"] = Array(std::vector<Json>{Json(Integer{static_cast<Integer::Int>(sizeof(T))})});
  column["version"] = 3;
  column["typestr"] = String(typestr);

  auto p_d_data = d_data.data().get();
  std::vector<Json> j_data{Json(Integer{reinterpret_cast<Integer::Int>(p_d_data)}),
                           Json(Boolean(false))};
  column["data"] = j_data;
  column["stream"] = nullptr;
  Json array(std::vector<Json>{column});

  std::string str;
  Json::Dump(array, &str);

  return str;
}

TEST(MetaInfo, FromInterface) {
  hipSetDevice(0);
  Context ctx;
  thrust::device_vector<float> d_data;

  std::string str = PrepareData<float>("<f4", &d_data);

  MetaInfo info;
  info.SetInfo(ctx, "label", str.c_str());

  auto const& h_label = info.labels.HostView();
  ASSERT_EQ(h_label.Size(), d_data.size());
  for (size_t i = 0; i < d_data.size(); ++i) {
    ASSERT_EQ(h_label(i), d_data[i]);
  }

  info.SetInfo(ctx, "weight", str.c_str());
  auto const& h_weight = info.weights_.HostVector();
  for (size_t i = 0; i < d_data.size(); ++i) {
    ASSERT_EQ(h_weight[i], d_data[i]);
  }

  info.SetInfo(ctx, "base_margin", str.c_str());
  auto const h_base_margin = info.base_margin_.View(GenericParameter::kCpuId);
  ASSERT_EQ(h_base_margin.Size(), d_data.size());
  for (size_t i = 0; i < d_data.size(); ++i) {
    ASSERT_EQ(h_base_margin(i), d_data[i]);
  }

  thrust::device_vector<int> d_group_data;
  std::string group_str = PrepareData<int>("<i4", &d_group_data, 4);
  d_group_data[0] = 4;
  d_group_data[1] = 3;
  d_group_data[2] = 2;
  d_group_data[3] = 1;
  info.SetInfo(ctx, "group", group_str.c_str());
  std::vector<bst_group_t> expected_group_ptr = {0, 4, 7, 9, 10};
  EXPECT_EQ(info.group_ptr_, expected_group_ptr);
}

TEST(MetaInfo, GPUStridedData) {
  TestMetaInfoStridedData(0);
}

TEST(MetaInfo, Group) {
  hipSetDevice(0);
  MetaInfo info;
  Context ctx;

  thrust::device_vector<uint32_t> d_uint;
  std::string uint_str = PrepareData<uint32_t>("<u4", &d_uint);
  info.SetInfo(ctx, "group", uint_str.c_str());
  auto& h_group = info.group_ptr_;
  ASSERT_EQ(h_group.size(), d_uint.size() + 1);
  for (size_t i = 1; i < h_group.size(); ++i) {
    ASSERT_EQ(h_group[i], d_uint[i - 1] + h_group[i - 1]) << "i: " << i;
  }

  thrust::device_vector<int64_t> d_int64;
  std::string int_str = PrepareData<int64_t>("<i8", &d_int64);
  info = MetaInfo();
  info.SetInfo(ctx, "group", int_str.c_str());
  h_group = info.group_ptr_;
  ASSERT_EQ(h_group.size(), d_uint.size() + 1);
  for (size_t i = 1; i < h_group.size(); ++i) {
    ASSERT_EQ(h_group[i], d_uint[i - 1] + h_group[i - 1]) << "i: " << i;
  }

  // Incorrect type
  thrust::device_vector<float> d_float;
  std::string float_str = PrepareData<float>("<f4", &d_float);
  info = MetaInfo();
  EXPECT_ANY_THROW(info.SetInfo(ctx, "group", float_str.c_str()));
}

TEST(MetaInfo, GPUQid) {
  xgboost::MetaInfo info;
  Context ctx;
  info.num_row_ = 100;
  thrust::device_vector<uint32_t> qid(info.num_row_, 0);
  for (size_t i = 0; i < qid.size(); ++i) {
    qid[i] = i;
  }
  auto column = Generate2dArrayInterface(info.num_row_, 1, "<u4", &qid);
  Json array{std::vector<Json>{column}};
  std::string array_str;
  Json::Dump(array, &array_str);
  info.SetInfo(ctx, "qid", array_str.c_str());
  ASSERT_EQ(info.group_ptr_.size(), info.num_row_ + 1);
  ASSERT_EQ(info.group_ptr_.front(), 0);
  ASSERT_EQ(info.group_ptr_.back(), info.num_row_);

  for (size_t i = 0; i < info.num_row_ + 1; ++i) {
    ASSERT_EQ(info.group_ptr_[i], i);
  }
}


TEST(MetaInfo, DeviceExtend) {
  dh::safe_cuda(hipSetDevice(0));
  size_t const kRows = 100;
  MetaInfo lhs, rhs;
  Context ctx;

  thrust::device_vector<float> d_data;
  std::string str = PrepareData<float>("<f4", &d_data, kRows);
  lhs.SetInfo(ctx, "label", str.c_str());
  rhs.SetInfo(ctx, "label", str.c_str());
  ASSERT_FALSE(rhs.labels.Data()->HostCanRead());
  lhs.num_row_ = kRows;
  rhs.num_row_ = kRows;

  lhs.Extend(rhs, true, true);
  ASSERT_EQ(lhs.num_row_, kRows * 2);
  ASSERT_FALSE(lhs.labels.Data()->HostCanRead());

  ASSERT_FALSE(lhs.labels.Data()->HostCanRead());
  ASSERT_FALSE(rhs.labels.Data()->HostCanRead());
}
}  // namespace xgboost
