/*! Copyright 2019 by Contributors */

#include <gtest/gtest.h>
#include <xgboost/data.h>
#include <xgboost/json.h>
#include <thrust/device_vector.h>
#include "../../../src/common/device_helpers.cuh"

namespace xgboost {

template <typename T>
std::string PrepareData(std::string typestr, thrust::device_vector<T>* out, const size_t kRows=16) {
  out->resize(kRows);
  auto& d_data = *out;

  for (size_t i = 0; i < d_data.size(); ++i) {
    d_data[i] = i * 2.0;
  }

  Json column { Object() };

  std::vector<Json> j_shape {Json(Integer(static_cast<Integer::Int>(kRows)))};
  column["shape"] = Array(j_shape);
  column["strides"] = Array(std::vector<Json>{Json(Integer(static_cast<Integer::Int>(sizeof(T))))});
  column["version"] = Integer(static_cast<Integer::Int>(1));
  column["typestr"] = String(typestr);

  auto p_d_data = d_data.data().get();
  std::vector<Json> j_data {
        Json(Integer(reinterpret_cast<Integer::Int>(p_d_data))),
        Json(Boolean(false))};
  column["data"] = j_data;
  Json array(std::vector<Json>{column});

  std::string str;
  Json::Dump(array, &str);

  return str;
}

TEST(MetaInfo, FromInterface) {
  hipSetDevice(0);
  thrust::device_vector<float> d_data;

  std::string str = PrepareData<float>("<f4", &d_data);

  MetaInfo info;
  info.SetInfo("label", str.c_str());

  auto const& h_label = info.labels_.HostVector();
  for (size_t i = 0; i < d_data.size(); ++i) {
    ASSERT_EQ(h_label[i], d_data[i]);
  }

  info.SetInfo("weight", str.c_str());
  auto const& h_weight = info.weights_.HostVector();
  for (size_t i = 0; i < d_data.size(); ++i) {
    ASSERT_EQ(h_weight[i], d_data[i]);
  }

  info.SetInfo("base_margin", str.c_str());
  auto const& h_base_margin = info.base_margin_.HostVector();
  for (size_t i = 0; i < d_data.size(); ++i) {
    ASSERT_EQ(h_base_margin[i], d_data[i]);
  }

  thrust::device_vector<int> d_group_data;
  std::string group_str = PrepareData<int>("<i4", &d_group_data, 4);
  d_group_data[0] = 4;
  d_group_data[1] = 3;
  d_group_data[2] = 2;
  d_group_data[3] = 1;
  info.SetInfo("group", group_str.c_str());
  std::vector<bst_group_t> expected_group_ptr = {0, 4, 7, 9, 10};
  EXPECT_EQ(info.group_ptr_, expected_group_ptr);
}

TEST(MetaInfo, Group) {
  hipSetDevice(0);
  MetaInfo info;

  thrust::device_vector<uint32_t> d_uint;
  std::string uint_str = PrepareData<uint32_t>("<u4", &d_uint);
  info.SetInfo("group", uint_str.c_str());
  auto& h_group = info.group_ptr_;
  ASSERT_EQ(h_group.size(), d_uint.size() + 1);
  for (size_t i = 1; i < h_group.size(); ++i) {
    ASSERT_EQ(h_group[i], d_uint[i - 1] + h_group[i - 1]) << "i: " << i;
  }

  thrust::device_vector<int64_t> d_int64;
  std::string int_str = PrepareData<int64_t>("<i8", &d_int64);
  info = MetaInfo();
  info.SetInfo("group", int_str.c_str());
  h_group = info.group_ptr_;
  ASSERT_EQ(h_group.size(), d_uint.size() + 1);
  for (size_t i = 1; i < h_group.size(); ++i) {
    ASSERT_EQ(h_group[i], d_uint[i - 1] + h_group[i - 1]) << "i: " << i;
  }

  // Incorrect type
  thrust::device_vector<float> d_float;
  std::string float_str = PrepareData<float>("<f4", &d_float);
  info = MetaInfo();
  EXPECT_ANY_THROW(info.SetInfo("group", float_str.c_str()));
}

TEST(MetaInfo, DeviceExtend) {
  dh::safe_cuda(hipSetDevice(0));
  size_t const kRows = 100;
  MetaInfo lhs, rhs;

  thrust::device_vector<float> d_data;
  std::string str = PrepareData<float>("<f4", &d_data, kRows);
  lhs.SetInfo("label", str.c_str());
  rhs.SetInfo("label", str.c_str());
  ASSERT_FALSE(rhs.labels_.HostCanRead());
  lhs.num_row_ = kRows;
  rhs.num_row_ = kRows;

  lhs.Extend(rhs, true);
  ASSERT_EQ(lhs.num_row_, kRows * 2);
  ASSERT_FALSE(lhs.labels_.HostCanRead());

  ASSERT_FALSE(lhs.labels_.HostCanRead());
  ASSERT_FALSE(rhs.labels_.HostCanRead());
}
}  // namespace xgboost
