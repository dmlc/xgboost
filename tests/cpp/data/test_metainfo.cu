/*! Copyright 2019 by Contributors */

#include <gtest/gtest.h>
#include <thrust/device_vector.h>
#include <xgboost/data.h>

#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/json_experimental.h"
#include "../../../src/common/json_reader_experimental.h"
#include "../../../src/common/json_writer_experimental.h"

namespace xgboost {

template <typename T>
std::string PrepareData(std::string typestr, thrust::device_vector<T>* out) {
  constexpr size_t kRows = 16;
  out->resize(kRows);
  auto& d_data = *out;

  for (size_t i = 0; i < d_data.size(); ++i) {
    d_data[i] = i * 2.0;
  }

  experimental::Document doc(experimental::ValueKind::kArray);

  {
    auto column = doc.GetValue().CreateArrayElem();
    column.SetObject();

    auto j_shape = column.CreateMember("shape");
    j_shape.SetArray().CreateArrayElem().SetInteger(kRows);

    auto j_strides = column.CreateMember("strides");
    j_strides.SetArray().CreateArrayElem().SetInteger(4);

    column.CreateMember("version").SetInteger(1);
    column.CreateMember("typestr").SetString(typestr);

    auto p_d_data = dh::Raw(d_data);
    auto j_data = column.CreateMember("data");
    j_data.SetArray();
    j_data.CreateArrayElem() = reinterpret_cast<int64_t>(p_d_data);
    j_data.CreateArrayElem().SetFalse();
  }

  std::string str = doc.Dump<experimental::JsonWriter>();
  return str;
}

TEST(MetaInfo, FromInterface) {
  hipSetDevice(0);
  thrust::device_vector<float> d_data;

  std::string str = PrepareData<float>("<f4", &d_data);

  MetaInfo info;
  info.SetInfo("label", str.c_str());

  auto const& h_label = info.labels_.HostVector();
  for (size_t i = 0; i < d_data.size(); ++i) {
    ASSERT_EQ(h_label[i], d_data[i]);
  }

  info.SetInfo("weight", str.c_str());
  auto const& h_weight = info.weights_.HostVector();
  for (size_t i = 0; i < d_data.size(); ++i) {
    ASSERT_EQ(h_weight[i], d_data[i]);
  }

  info.SetInfo("base_margin", str.c_str());
  auto const& h_base_margin = info.base_margin_.HostVector();
  for (size_t i = 0; i < d_data.size(); ++i) {
    ASSERT_EQ(h_base_margin[i], d_data[i]);
  }

  EXPECT_ANY_THROW({info.SetInfo("group", str.c_str());});
}

TEST(MetaInfo, Group) {
  hipSetDevice(0);
  thrust::device_vector<uint32_t> d_data;
  std::string str = PrepareData<uint32_t>("<u4", &d_data);

  MetaInfo info;

  info.SetInfo("group", str.c_str());
  auto const& h_group = info.group_ptr_;
  ASSERT_EQ(h_group.size(), d_data.size() + 1);
  for (size_t i = 1; i < h_group.size(); ++i) {
    ASSERT_EQ(h_group[i], d_data[i-1] + h_group[i-1]) << "i: " << i;
  }
}
}  // namespace xgboost