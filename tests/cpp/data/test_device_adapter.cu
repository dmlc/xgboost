// Copyright (c) 2019 by Contributors
#include <gtest/gtest.h>
#include <xgboost/data.h>
#include "../../../src/data/adapter.h"
#include "../../../src/data/simple_dmatrix.h"
#include "../../../src/common/timer.h"
#include "../helpers.h"
#include <thrust/device_vector.h>
#include "../../../src/data/device_adapter.cuh"
#include "test_array_interface.h"
using namespace xgboost;  // NOLINT

void TestCudfAdapter()
{
  constexpr size_t kRowsA {16};
  constexpr size_t kRowsB {16};
  std::vector<Json> columns;
  thrust::device_vector<double> d_data_0(kRowsA);
  thrust::device_vector<uint32_t> d_data_1(kRowsB);

  columns.emplace_back(GenerateDenseColumn<double>("<f8", kRowsA, &d_data_0));
  columns.emplace_back(GenerateDenseColumn<uint32_t>("<u4", kRowsB, &d_data_1));

  Json column_arr {columns};

  std::string str;
  Json::Dump(column_arr, &str);

  data::CudfAdapter adapter(str);

  adapter.Next();
  auto & batch = adapter.Value();
  EXPECT_EQ(batch.Size(), kRowsA + kRowsB);

  EXPECT_NO_THROW({
    dh::LaunchN(batch.Size(), [=] __device__(size_t idx) {
      auto element = batch.GetElement(idx);
      KERNEL_CHECK(element.row_idx == idx / 2);
      if (idx % 2 == 0) {
        KERNEL_CHECK(element.column_idx == 0);
        KERNEL_CHECK(element.value == element.row_idx * 2.0f);
      } else {
        KERNEL_CHECK(element.column_idx == 1);
        KERNEL_CHECK(element.value == element.row_idx * 2.0f);
      }
    });
    dh::safe_cuda(hipDeviceSynchronize());
  });
}

TEST(DeviceAdapter, CudfAdapter) {
  TestCudfAdapter();
}

namespace xgboost::data {
TEST(DeviceAdapter, GetRowCounts) {
  auto ctx = MakeCUDACtx(0);

  for (bst_feature_t n_features : {1, 2, 4, 64, 128, 256}) {
    HostDeviceVector<float> storage;
    auto str_arr = RandomDataGenerator{8192, n_features, 0.0}
                       .Device(ctx.Device())
                       .GenerateArrayInterface(&storage);
    auto adapter = CupyAdapter{str_arr};
    HostDeviceVector<bst_row_t> offset(adapter.NumRows() + 1, 0);
    offset.SetDevice(ctx.Device());
    auto rstride = GetRowCounts(adapter.Value(), offset.DeviceSpan(), ctx.Device(),
                                std::numeric_limits<float>::quiet_NaN());
    ASSERT_EQ(rstride, n_features);
  }
}
}  // namespace xgboost::data
