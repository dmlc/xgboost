// Copyright (c) 2019 by Contributors
#include <gtest/gtest.h>
#include <xgboost/data.h>
#include "../../../src/data/adapter.h"
#include "../../../src/data/simple_dmatrix.h"
#include "../../../src/common/timer.h"
#include "../helpers.h"
#include <thrust/device_vector.h>
#include "../../../src/data/device_adapter.cuh"
#include "test_array_interface.h"
using namespace xgboost;  // NOLINT

void TestCudfAdapter()
{
  constexpr size_t kRowsA {16};
  constexpr size_t kRowsB {16};
  std::vector<Json> columns;
  thrust::device_vector<double> d_data_0(kRowsA);
  thrust::device_vector<uint32_t> d_data_1(kRowsB);

  columns.emplace_back(GenerateDenseColumn<double>("<f8", kRowsA, &d_data_0));
  columns.emplace_back(GenerateDenseColumn<uint32_t>("<u4", kRowsB, &d_data_1));

  Json column_arr {columns};

  std::stringstream ss;
  Json::Dump(column_arr, &ss);
  std::string str = ss.str();

  data::CudfAdapter adapter(str);

  adapter.Next();
  auto & batch = adapter.Value();
  EXPECT_EQ(batch.Size(), kRowsA + kRowsB);

  EXPECT_NO_THROW({
    dh::LaunchN(0, batch.Size(), [=] __device__(size_t idx) {
      auto element = batch.GetElement(idx);
      if (idx < kRowsA) {
        KERNEL_CHECK(element.column_idx == 0);
        KERNEL_CHECK(element.row_idx == idx);
        KERNEL_CHECK(element.value == element.row_idx * 2.0f);
      } else {
        KERNEL_CHECK(element.column_idx == 1);
        KERNEL_CHECK(element.row_idx == idx - kRowsA);
        KERNEL_CHECK(element.value == element.row_idx * 2.0f);
      }
    });
    dh::safe_cuda(hipDeviceSynchronize());
  });
}

TEST(device_adapter, CudfAdapter) {
  TestCudfAdapter();
}
