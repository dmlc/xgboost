#include "hip/hip_runtime.h"
// Copyright by Contributors

#include "../../../src/common/compressed_iterator.h"
#include "../../../src/data/ellpack_page.cuh"
#include "../../../src/data/sparse_page_dmatrix.h"
#include "../filesystem.h"  // dmlc::TemporaryDirectory
#include "../helpers.h"

namespace xgboost {

TEST(SparsePageDMatrix, EllpackPage) {
  dmlc::TemporaryDirectory tempdir;
  const std::string tmp_file = tempdir.path + "/simple.libsvm";
  CreateSimpleTestData(tmp_file);
  DMatrix* dmat = DMatrix::Load(tmp_file + "#" + tmp_file + ".cache", true, false);

  // Loop over the batches and assert the data is as expected
  size_t n = 0;
  for (const auto& batch : dmat->GetBatches<EllpackPage>({0, 256})) {
    n += batch.Size();
  }
  EXPECT_EQ(n, dmat->Info().num_row_);

  auto path =
      data::MakeId(tmp_file + ".cache",
                   dynamic_cast<data::SparsePageDMatrix *>(dmat)) +
      ".row.page";
  EXPECT_TRUE(FileExists(path));
  path =
      data::MakeId(tmp_file + ".cache",
                   dynamic_cast<data::SparsePageDMatrix *>(dmat)) +
      ".ellpack.page";
  EXPECT_TRUE(FileExists(path));

  delete dmat;
}

TEST(SparsePageDMatrix, MultipleEllpackPages) {
  dmlc::TemporaryDirectory tmpdir;
  std::string filename = tmpdir.path + "/big.libsvm";
  size_t constexpr kPageSize = 64, kEntriesPerCol = 3;
  size_t constexpr kEntries = kPageSize * kEntriesPerCol * 2;
  std::unique_ptr<DMatrix> dmat = CreateSparsePageDMatrix(kEntries, filename);

  // Loop over the batches and count the records
  int64_t batch_count = 0;
  int64_t row_count = 0;
  for (const auto& batch : dmat->GetBatches<EllpackPage>({0, 256})) {
    EXPECT_LT(batch.Size(), dmat->Info().num_row_);
    batch_count++;
    row_count += batch.Size();
  }
  EXPECT_GE(batch_count, 2);
  EXPECT_EQ(row_count, dmat->Info().num_row_);

  auto path =
      data::MakeId(filename,
                   dynamic_cast<data::SparsePageDMatrix *>(dmat.get())) +
      ".ellpack.page";
}

TEST(SparsePageDMatrix, RetainEllpackPage) {
  auto m = CreateSparsePageDMatrix(10000);
  auto batches = m->GetBatches<EllpackPage>({0, 32});
  auto begin = batches.begin();
  auto end = batches.end();

  std::vector<HostDeviceVector<common::CompressedByteT>> gidx_buffers;
  std::vector<std::shared_ptr<EllpackPage const>> iterators;
  for (auto it = begin; it != end; ++it) {
    iterators.push_back(it.Page());
    gidx_buffers.emplace_back(HostDeviceVector<common::CompressedByteT>{});
    gidx_buffers.back().Resize((*it).Impl()->gidx_buffer.Size());
    gidx_buffers.back().Copy((*it).Impl()->gidx_buffer);
  }
  ASSERT_GE(iterators.size(), 2);

  for (size_t i = 0; i < iterators.size(); ++i) {
    ASSERT_EQ((*iterators[i]).Impl()->gidx_buffer.HostVector(), gidx_buffers.at(i).HostVector());
    if (i != iterators.size() - 1) {
      ASSERT_EQ(iterators[i].use_count(), 1);
    } else {
      // The last batch is still being held by sparse page DMatrix.
      ASSERT_EQ(iterators[i].use_count(), 2);
    }
  }

  // make sure it's const and the caller can not modify the content of page.
  for (auto& page : m->GetBatches<EllpackPage>({0, 32})) {
    static_assert(std::is_const<std::remove_reference_t<decltype(page)>>::value, "");
  }

  // The above iteration clears out all references inside DMatrix.
  for (auto const& ptr : iterators) {
    ASSERT_TRUE(ptr.unique());
  }
}

TEST(SparsePageDMatrix, EllpackPageContent) {
  constexpr size_t kRows = 6;
  constexpr size_t kCols = 2;
  constexpr size_t kPageSize = 1;

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  // Create a DMatrix with multiple batches.
  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix>
      dmat_ext(CreateSparsePageDMatrixWithRC(kRows, kCols, kPageSize, true, tmpdir));

  BatchParam param{0, 2};
  auto impl = (*dmat->GetBatches<EllpackPage>(param).begin()).Impl();
  EXPECT_EQ(impl->base_rowid, 0);
  EXPECT_EQ(impl->n_rows, kRows);
  EXPECT_FALSE(impl->is_dense);
  EXPECT_EQ(impl->row_stride, 2);
  EXPECT_EQ(impl->Cuts().TotalBins(), 4);

  std::unique_ptr<EllpackPageImpl> impl_ext;
  size_t offset = 0;
  for (auto& batch : dmat_ext->GetBatches<EllpackPage>(param)) {
    if (!impl_ext) {
      impl_ext.reset(new EllpackPageImpl(
          batch.Impl()->gidx_buffer.DeviceIdx(), batch.Impl()->Cuts(),
          batch.Impl()->is_dense, batch.Impl()->row_stride, kRows));
    }
    auto n_elems = impl_ext->Copy(0, batch.Impl(), offset);
    offset += n_elems;
  }
  EXPECT_EQ(impl_ext->base_rowid, 0);
  EXPECT_EQ(impl_ext->n_rows, kRows);
  EXPECT_FALSE(impl_ext->is_dense);
  EXPECT_EQ(impl_ext->row_stride, 2);
  EXPECT_EQ(impl_ext->Cuts().TotalBins(), 4);

  std::vector<common::CompressedByteT> buffer(impl->gidx_buffer.HostVector());
  std::vector<common::CompressedByteT> buffer_ext(impl_ext->gidx_buffer.HostVector());
  EXPECT_EQ(buffer, buffer_ext);
}

struct ReadRowFunction {
  EllpackDeviceAccessor matrix;
  int row;
  bst_float* row_data_d;
  ReadRowFunction(EllpackDeviceAccessor matrix, int row, bst_float* row_data_d)
      : matrix(std::move(matrix)), row(row), row_data_d(row_data_d) {}

  __device__ void operator()(size_t col) {
    auto value = matrix.GetFvalue(row, col);
    if (isnan(value)) {
      value = -1;
    }
    row_data_d[col] = value;
  }
};

TEST(SparsePageDMatrix, MultipleEllpackPageContent) {
  constexpr size_t kRows = 6;
  constexpr size_t kCols = 2;
  constexpr int kMaxBins = 256;
  constexpr size_t kPageSize = 1;

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  // Create a DMatrix with multiple batches.
  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix>
      dmat_ext(CreateSparsePageDMatrixWithRC(kRows, kCols, kPageSize, true, tmpdir));

  BatchParam param{0, kMaxBins};
  auto impl = (*dmat->GetBatches<EllpackPage>(param).begin()).Impl();
  EXPECT_EQ(impl->base_rowid, 0);
  EXPECT_EQ(impl->n_rows, kRows);

  size_t current_row = 0;
  thrust::device_vector<bst_float> row_d(kCols);
  thrust::device_vector<bst_float> row_ext_d(kCols);
  std::vector<bst_float> row(kCols);
  std::vector<bst_float> row_ext(kCols);
  for (auto& page : dmat_ext->GetBatches<EllpackPage>(param)) {
    auto impl_ext = page.Impl();
    EXPECT_EQ(impl_ext->base_rowid, current_row);

    for (size_t i = 0; i < impl_ext->Size(); i++) {
      dh::LaunchN(kCols, ReadRowFunction(impl->GetDeviceAccessor(0), current_row, row_d.data().get()));
      thrust::copy(row_d.begin(), row_d.end(), row.begin());

      dh::LaunchN(kCols, ReadRowFunction(impl_ext->GetDeviceAccessor(0), current_row, row_ext_d.data().get()));
      thrust::copy(row_ext_d.begin(), row_ext_d.end(), row_ext.begin());

      EXPECT_EQ(row, row_ext);
      current_row++;
    }
  }
}

TEST(SparsePageDMatrix, EllpackPageMultipleLoops) {
  constexpr size_t kRows = 1024;
  constexpr size_t kCols = 16;
  constexpr int kMaxBins = 256;
  constexpr size_t kPageSize = 4096;

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  // Create a DMatrix with multiple batches.
  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix>
      dmat_ext(CreateSparsePageDMatrixWithRC(kRows, kCols, kPageSize, true, tmpdir));

  BatchParam param{0, kMaxBins};

  size_t current_row = 0;
  for (auto& page : dmat_ext->GetBatches<EllpackPage>(param)) {
    auto impl_ext = page.Impl();
    EXPECT_EQ(impl_ext->base_rowid, current_row);
    current_row += impl_ext->n_rows;
  }
}

}  // namespace xgboost
