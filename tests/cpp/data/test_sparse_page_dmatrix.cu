#include "hip/hip_runtime.h"
/**
 * Copyright 2019-2023 by XGBoost Contributors
 */
#include <xgboost/data.h>  // for DMatrix

#include "../../../src/common/compressed_iterator.h"
#include "../../../src/data/ellpack_page.cuh"
#include "../../../src/data/ellpack_page.h"
#include "../../../src/data/sparse_page_dmatrix.h"
#include "../../../src/tree/param.h"  // TrainParam
#include "../filesystem.h"            // dmlc::TemporaryDirectory
#include "../helpers.h"

namespace xgboost {

TEST(SparsePageDMatrix, EllpackPage) {
  Context ctx{MakeCUDACtx(0)};
  auto param = BatchParam{256, tree::TrainParam::DftSparseThreshold()};
  dmlc::TemporaryDirectory tempdir;
  const std::string tmp_file = tempdir.path + "/simple.libsvm";
  CreateSimpleTestData(tmp_file);
  DMatrix* dmat = DMatrix::Load(tmp_file + "?format=libsvm" + "#" + tmp_file + ".cache");

  // Loop over the batches and assert the data is as expected
  size_t n = 0;
  for (const auto& batch : dmat->GetBatches<EllpackPage>(&ctx, param)) {
    n += batch.Size();
  }
  EXPECT_EQ(n, dmat->Info().num_row_);

  auto path =
      data::MakeId(tmp_file + ".cache",
                   dynamic_cast<data::SparsePageDMatrix *>(dmat)) +
      ".row.page";
  EXPECT_TRUE(FileExists(path));
  path =
      data::MakeId(tmp_file + ".cache",
                   dynamic_cast<data::SparsePageDMatrix *>(dmat)) +
      ".ellpack.page";
  EXPECT_TRUE(FileExists(path));

  delete dmat;
}

TEST(SparsePageDMatrix, MultipleEllpackPages) {
  Context ctx{MakeCUDACtx(0)};
  auto param = BatchParam{256, tree::TrainParam::DftSparseThreshold()};
  dmlc::TemporaryDirectory tmpdir;
  std::string filename = tmpdir.path + "/big.libsvm";
  size_t constexpr kPageSize = 64, kEntriesPerCol = 3;
  size_t constexpr kEntries = kPageSize * kEntriesPerCol * 2;
  std::unique_ptr<DMatrix> dmat = CreateSparsePageDMatrix(kEntries, filename);

  // Loop over the batches and count the records
  int64_t batch_count = 0;
  int64_t row_count = 0;
  for (const auto& batch : dmat->GetBatches<EllpackPage>(&ctx, param)) {
    EXPECT_LT(batch.Size(), dmat->Info().num_row_);
    batch_count++;
    row_count += batch.Size();
  }
  EXPECT_GE(batch_count, 2);
  EXPECT_EQ(row_count, dmat->Info().num_row_);

  auto path =
      data::MakeId(filename,
                   dynamic_cast<data::SparsePageDMatrix *>(dmat.get())) +
      ".ellpack.page";
}

TEST(SparsePageDMatrix, RetainEllpackPage) {
  Context ctx{MakeCUDACtx(0)};
  auto param = BatchParam{32, tree::TrainParam::DftSparseThreshold()};
  auto m = CreateSparsePageDMatrix(10000);

  auto batches = m->GetBatches<EllpackPage>(&ctx, param);
  auto begin = batches.begin();
  auto end = batches.end();

  std::vector<HostDeviceVector<common::CompressedByteT>> gidx_buffers;
  std::vector<std::shared_ptr<EllpackPage const>> iterators;
  for (auto it = begin; it != end; ++it) {
    iterators.push_back(it.Page());
    gidx_buffers.emplace_back();
    gidx_buffers.back().Resize((*it).Impl()->gidx_buffer.Size());
    gidx_buffers.back().Copy((*it).Impl()->gidx_buffer);
  }
  ASSERT_GE(iterators.size(), 2);

  for (size_t i = 0; i < iterators.size(); ++i) {
    ASSERT_EQ((*iterators[i]).Impl()->gidx_buffer.HostVector(), gidx_buffers.at(i).HostVector());
    if (i != iterators.size() - 1) {
      ASSERT_EQ(iterators[i].use_count(), 1);
    } else {
      // The last batch is still being held by sparse page DMatrix.
      ASSERT_EQ(iterators[i].use_count(), 2);
    }
  }

  // make sure it's const and the caller can not modify the content of page.
  for (auto& page : m->GetBatches<EllpackPage>(&ctx, param)) {
    static_assert(std::is_const<std::remove_reference_t<decltype(page)>>::value);
  }

  // The above iteration clears out all references inside DMatrix.
  for (auto const& ptr : iterators) {
    ASSERT_TRUE(ptr.unique());
  }
}

TEST(SparsePageDMatrix, EllpackPageContent) {
  auto ctx = MakeCUDACtx(0);
  constexpr size_t kRows = 6;
  constexpr size_t kCols = 2;
  constexpr size_t kPageSize = 1;

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  // Create a DMatrix with multiple batches.
  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix>
      dmat_ext(CreateSparsePageDMatrixWithRC(kRows, kCols, kPageSize, true, tmpdir));

  auto param = BatchParam{2, tree::TrainParam::DftSparseThreshold()};
  auto impl = (*dmat->GetBatches<EllpackPage>(&ctx, param).begin()).Impl();
  EXPECT_EQ(impl->base_rowid, 0);
  EXPECT_EQ(impl->n_rows, kRows);
  EXPECT_FALSE(impl->is_dense);
  EXPECT_EQ(impl->row_stride, 2);
  EXPECT_EQ(impl->Cuts().TotalBins(), 4);

  std::unique_ptr<EllpackPageImpl> impl_ext;
  size_t offset = 0;
  for (auto& batch : dmat_ext->GetBatches<EllpackPage>(&ctx, param)) {
    if (!impl_ext) {
      impl_ext = std::make_unique<EllpackPageImpl>(batch.Impl()->gidx_buffer.Device(),
                                                   batch.Impl()->Cuts(), batch.Impl()->is_dense,
                                                   batch.Impl()->row_stride, kRows);
    }
    auto n_elems = impl_ext->Copy(ctx.Device(), batch.Impl(), offset);
    offset += n_elems;
  }
  EXPECT_EQ(impl_ext->base_rowid, 0);
  EXPECT_EQ(impl_ext->n_rows, kRows);
  EXPECT_FALSE(impl_ext->is_dense);
  EXPECT_EQ(impl_ext->row_stride, 2);
  EXPECT_EQ(impl_ext->Cuts().TotalBins(), 4);

  std::vector<common::CompressedByteT> buffer(impl->gidx_buffer.HostVector());
  std::vector<common::CompressedByteT> buffer_ext(impl_ext->gidx_buffer.HostVector());
  EXPECT_EQ(buffer, buffer_ext);
}

struct ReadRowFunction {
  EllpackDeviceAccessor matrix;
  int row;
  bst_float* row_data_d;
  ReadRowFunction(EllpackDeviceAccessor matrix, int row, bst_float* row_data_d)
      : matrix(std::move(matrix)), row(row), row_data_d(row_data_d) {}

  __device__ void operator()(size_t col) {
    auto value = matrix.GetFvalue(row, col);
    if (isnan(value)) {
      value = -1;
    }
    row_data_d[col] = value;
  }
};

TEST(SparsePageDMatrix, MultipleEllpackPageContent) {
  constexpr size_t kRows = 6;
  constexpr size_t kCols = 2;
  constexpr int kMaxBins = 256;
  constexpr size_t kPageSize = 1;

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  // Create a DMatrix with multiple batches.
  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix>
      dmat_ext(CreateSparsePageDMatrixWithRC(kRows, kCols, kPageSize, true, tmpdir));

  Context ctx{MakeCUDACtx(0)};
  auto param = BatchParam{kMaxBins, tree::TrainParam::DftSparseThreshold()};
  auto impl = (*dmat->GetBatches<EllpackPage>(&ctx, param).begin()).Impl();
  EXPECT_EQ(impl->base_rowid, 0);
  EXPECT_EQ(impl->n_rows, kRows);

  size_t current_row = 0;
  thrust::device_vector<bst_float> row_d(kCols);
  thrust::device_vector<bst_float> row_ext_d(kCols);
  std::vector<bst_float> row(kCols);
  std::vector<bst_float> row_ext(kCols);
  for (auto& page : dmat_ext->GetBatches<EllpackPage>(&ctx, param)) {
    auto impl_ext = page.Impl();
    EXPECT_EQ(impl_ext->base_rowid, current_row);

    for (size_t i = 0; i < impl_ext->Size(); i++) {
      dh::LaunchN(kCols, ReadRowFunction(impl->GetDeviceAccessor(ctx.Device()), current_row,
                                         row_d.data().get()));
      thrust::copy(row_d.begin(), row_d.end(), row.begin());

      dh::LaunchN(kCols, ReadRowFunction(impl_ext->GetDeviceAccessor(ctx.Device()), current_row,
                                         row_ext_d.data().get()));
      thrust::copy(row_ext_d.begin(), row_ext_d.end(), row_ext.begin());

      EXPECT_EQ(row, row_ext);
      current_row++;
    }
  }
}

TEST(SparsePageDMatrix, EllpackPageMultipleLoops) {
  constexpr size_t kRows = 1024;
  constexpr size_t kCols = 16;
  constexpr int kMaxBins = 256;
  constexpr size_t kPageSize = 4096;

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  // Create a DMatrix with multiple batches.
  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix>
      dmat_ext(CreateSparsePageDMatrixWithRC(kRows, kCols, kPageSize, true, tmpdir));

  Context ctx{MakeCUDACtx(0)};
  auto param = BatchParam{kMaxBins, tree::TrainParam::DftSparseThreshold()};

  size_t current_row = 0;
  for (auto& page : dmat_ext->GetBatches<EllpackPage>(&ctx, param)) {
    auto impl_ext = page.Impl();
    EXPECT_EQ(impl_ext->base_rowid, current_row);
    current_row += impl_ext->n_rows;
  }
}

}  // namespace xgboost
