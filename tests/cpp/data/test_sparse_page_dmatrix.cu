#include "hip/hip_runtime.h"
// Copyright by Contributors

#include <dmlc/filesystem.h>
#include "../helpers.h"
#include "../../../src/common/compressed_iterator.h"

namespace xgboost {

TEST(SparsePageDMatrix, EllpackPage) {
  dmlc::TemporaryDirectory tempdir;
  const std::string tmp_file = tempdir.path + "/simple.libsvm";
  CreateSimpleTestData(tmp_file);
  DMatrix* dmat = DMatrix::Load(tmp_file + "#" + tmp_file + ".cache", true, false);

  // Loop over the batches and assert the data is as expected
  for (const auto& batch : dmat->GetBatches<EllpackPage>({0, 256, 64})) {
    EXPECT_EQ(batch.Size(), dmat->Info().num_row_);
  }

  EXPECT_TRUE(FileExists(tmp_file + ".cache"));
  EXPECT_TRUE(FileExists(tmp_file + ".cache.row.page"));
  EXPECT_TRUE(FileExists(tmp_file + ".cache.ellpack.page"));

  delete dmat;
}

TEST(SparsePageDMatrix, MultipleEllpackPages) {
  dmlc::TemporaryDirectory tmpdir;
  std::string filename = tmpdir.path + "/big.libsvm";
  std::unique_ptr<DMatrix> dmat = CreateSparsePageDMatrix(12, 64, filename);

  // Loop over the batches and count the records
  int64_t batch_count = 0;
  int64_t row_count = 0;
  for (const auto& batch : dmat->GetBatches<EllpackPage>({0, 256, 0, 7UL})) {
    EXPECT_LT(batch.Size(), dmat->Info().num_row_);
    batch_count++;
    row_count += batch.Size();
  }
  EXPECT_GE(batch_count, 2);
  EXPECT_EQ(row_count, dmat->Info().num_row_);

  EXPECT_TRUE(FileExists(filename + ".cache.ellpack.page"));
}

TEST(SparsePageDMatrix, EllpackPageContent) {
  constexpr size_t kRows = 6;
  constexpr size_t kCols = 2;
  constexpr size_t kPageSize = 1;

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  // Create a DMatrix with multiple batches.
  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix>
      dmat_ext(CreateSparsePageDMatrixWithRC(kRows, kCols, kPageSize, true, tmpdir));

  BatchParam param{0, 2, 0, 0};
  auto impl = (*dmat->GetBatches<EllpackPage>(param).begin()).Impl();
  EXPECT_EQ(impl->matrix.base_rowid, 0);
  EXPECT_EQ(impl->matrix.n_rows, kRows);
  EXPECT_FALSE(impl->matrix.info.is_dense);
  EXPECT_EQ(impl->matrix.info.row_stride, 2);
  EXPECT_EQ(impl->matrix.info.n_bins, 4);

  auto impl_ext = (*dmat_ext->GetBatches<EllpackPage>(param).begin()).Impl();
  EXPECT_EQ(impl_ext->matrix.base_rowid, 0);
  EXPECT_EQ(impl_ext->matrix.n_rows, kRows);
  EXPECT_FALSE(impl_ext->matrix.info.is_dense);
  EXPECT_EQ(impl_ext->matrix.info.row_stride, 2);
  EXPECT_EQ(impl_ext->matrix.info.n_bins, 4);

  std::vector<common::CompressedByteT> buffer(impl->gidx_buffer.size());
  std::vector<common::CompressedByteT> buffer_ext(impl_ext->gidx_buffer.size());
  dh::CopyDeviceSpanToVector(&buffer, impl->gidx_buffer);
  dh::CopyDeviceSpanToVector(&buffer_ext, impl_ext->gidx_buffer);
  EXPECT_EQ(buffer, buffer_ext);
}

struct ReadRowFunction {
  EllpackMatrix matrix;
  int row;
  bst_float* row_data_d;
  ReadRowFunction(EllpackMatrix matrix, int row, bst_float* row_data_d)
      : matrix(std::move(matrix)), row(row), row_data_d(row_data_d) {}

  __device__ void operator()(size_t col) {
    auto value = matrix.GetElement(row, col);
    if (isnan(value)) {
      value = -1;
    }
    row_data_d[col] = value;
  }
};

TEST(SparsePageDMatrix, MultipleEllpackPageContent) {
  constexpr size_t kRows = 6;
  constexpr size_t kCols = 2;
  constexpr int kMaxBins = 256;
  constexpr size_t kPageSize = 1;

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  // Create a DMatrix with multiple batches.
  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix>
      dmat_ext(CreateSparsePageDMatrixWithRC(kRows, kCols, kPageSize, true, tmpdir));

  BatchParam param{0, kMaxBins, 0, kPageSize};
  auto impl = (*dmat->GetBatches<EllpackPage>(param).begin()).Impl();
  EXPECT_EQ(impl->matrix.base_rowid, 0);
  EXPECT_EQ(impl->matrix.n_rows, kRows);

  size_t current_row = 0;
  thrust::device_vector<bst_float> row_d(kCols);
  thrust::device_vector<bst_float> row_ext_d(kCols);
  std::vector<bst_float> row(kCols);
  std::vector<bst_float> row_ext(kCols);
  for (auto& page : dmat_ext->GetBatches<EllpackPage>(param)) {
    auto impl_ext = page.Impl();
    EXPECT_EQ(impl_ext->matrix.base_rowid, current_row);

    for (size_t i = 0; i < impl_ext->Size(); i++) {
      dh::LaunchN(0, kCols, ReadRowFunction(impl->matrix, current_row, row_d.data().get()));
      thrust::copy(row_d.begin(), row_d.end(), row.begin());

      dh::LaunchN(0, kCols, ReadRowFunction(impl_ext->matrix, current_row, row_ext_d.data().get()));
      thrust::copy(row_ext_d.begin(), row_ext_d.end(), row_ext.begin());

      EXPECT_EQ(row, row_ext);
      current_row++;
    }
  }
}

TEST(SparsePageDMatrix, EllpackPageMultipleLoops) {
  constexpr size_t kRows = 1024;
  constexpr size_t kCols = 16;
  constexpr int kMaxBins = 256;
  constexpr size_t kPageSize = 4096;

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  // Create a DMatrix with multiple batches.
  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix>
      dmat_ext(CreateSparsePageDMatrixWithRC(kRows, kCols, kPageSize, true, tmpdir));

  BatchParam param{0, kMaxBins, 0, kPageSize};
  auto impl = (*dmat->GetBatches<EllpackPage>(param).begin()).Impl();

  size_t current_row = 0;
  for (auto& page : dmat_ext->GetBatches<EllpackPage>(param)) {
    auto impl_ext = page.Impl();
    EXPECT_EQ(impl_ext->matrix.base_rowid, current_row);
    current_row += impl_ext->matrix.n_rows;
  }

  current_row = 0;
  thrust::device_vector<bst_float> row_d(kCols);
  thrust::device_vector<bst_float> row_ext_d(kCols);
  std::vector<bst_float> row(kCols);
  std::vector<bst_float> row_ext(kCols);
  for (auto& page : dmat_ext->GetBatches<EllpackPage>(param)) {
    auto impl_ext = page.Impl();
    EXPECT_EQ(impl_ext->matrix.base_rowid, current_row);

    for (size_t i = 0; i < impl_ext->Size(); i++) {
      dh::LaunchN(0, kCols, ReadRowFunction(impl->matrix, current_row, row_d.data().get()));
      thrust::copy(row_d.begin(), row_d.end(), row.begin());

      dh::LaunchN(0, kCols, ReadRowFunction(impl_ext->matrix, current_row, row_ext_d.data().get()));
      thrust::copy(row_ext_d.begin(), row_ext_d.end(), row_ext.begin());

      EXPECT_EQ(row, row_ext) << "for row " << current_row;

      current_row++;
    }
  }
}

}  // namespace xgboost
