#include "hip/hip_runtime.h"
// Copyright (c) 2019 by Contributors
#include <gtest/gtest.h>
#include <xgboost/data.h>
#include <xgboost/json.h>
#include <thrust/device_vector.h>

#include <memory>
#include "../../../src/common/bitfield.h"
#include "../../../src/common/device_helpers.cuh"
#include "../../../src/data/simple_csr_source.h"
#include "../../../src/data/columnar.h"

namespace xgboost {

TEST(ArrayInterfaceHandler, Error) {
  constexpr size_t kRows {16};
  Json column { Object() };
  std::vector<Json> j_shape {Json(Integer(static_cast<Integer::Int>(kRows)))};
  column["shape"] = Array(j_shape);
  std::vector<Json> j_data {
    Json(Integer(reinterpret_cast<Integer::Int>(nullptr))),
        Json(Boolean(false))};

  auto const& column_obj = get<Object>(column);
  // missing version
  EXPECT_THROW(Columnar c(column_obj), dmlc::Error);
  column["version"] = Integer(static_cast<Integer::Int>(1));
  // missing data
  EXPECT_THROW(Columnar c(column_obj), dmlc::Error);
  column["data"] = j_data;
  // missing typestr
  EXPECT_THROW(Columnar c(column_obj), dmlc::Error);
  column["typestr"] = String("<f4");
  // nullptr is not valid
  EXPECT_THROW(Columnar c(column_obj), dmlc::Error);
  thrust::device_vector<float> d_data(kRows);
  j_data = {Json(Integer(reinterpret_cast<Integer::Int>(d_data.data().get()))),
            Json(Boolean(false))};
  column["data"] = j_data;
  EXPECT_NO_THROW(Columnar c(column_obj));

  std::vector<Json> j_mask_shape {Json(Integer(static_cast<Integer::Int>(kRows - 1)))};
  column["mask"] = Object();
  column["mask"]["shape"] = j_mask_shape;
  column["mask"]["data"] = j_data;
  column["mask"]["typestr"] = String("<i1");
  column["mask"]["version"] = Integer(static_cast<Integer::Int>(1));
  // shape of mask and data doesn't match.
  EXPECT_THROW(Columnar c(column_obj), dmlc::Error);
}

template <typename T>
Json GenerateDenseColumn(std::string const& typestr, size_t kRows,
                         thrust::device_vector<T>* out_d_data) {
  auto& d_data = *out_d_data;
  Json column { Object() };
  std::vector<Json> j_shape {Json(Integer(static_cast<Integer::Int>(kRows)))};
  column["shape"] = Array(j_shape);
  column["strides"] = Array(std::vector<Json>{Json(Integer(static_cast<Integer::Int>(sizeof(T))))});

  d_data.resize(kRows);
  for (size_t i = 0; i < d_data.size(); ++i) {
    d_data[i] = i * 2.0;
  }

  auto p_d_data = dh::Raw(d_data);

  std::vector<Json> j_data {
    Json(Integer(reinterpret_cast<Integer::Int>(p_d_data))),
        Json(Boolean(false))};
  column["data"] = j_data;

  column["version"] = Integer(static_cast<Integer::Int>(1));
  column["typestr"] = String(typestr);
  return column;
}

void TestGetElement() {
  thrust::device_vector<float> data;
  auto j_column = GenerateDenseColumn("<f4", 3, &data);
  auto const& column_obj = get<Object const>(j_column);
  Columnar foreign_column(column_obj);

  EXPECT_NO_THROW({
    dh::LaunchN(0, 1, [=] __device__(size_t idx) {
      KERNEL_CHECK(foreign_column.GetElement(0) == 0.0f);
      KERNEL_CHECK(foreign_column.GetElement(1) == 2.0f);
      KERNEL_CHECK(foreign_column.GetElement(2) == 4.0f);
    });
  });
}

TEST(Columnar, GetElement) { TestGetElement(); }

void TestDenseColumn(std::unique_ptr<data::SimpleCSRSource> const& source,
                     size_t n_rows, size_t n_cols) {
  auto const& data = source->page_.data.HostVector();
  auto const& offset = source->page_.offset.HostVector();

  for (size_t i = 0; i < n_rows; i++) {
    auto const idx = i * n_cols;
    auto const e_0 = data.at(idx);
    ASSERT_NEAR(e_0.fvalue, i * 2.0, kRtEps) << "idx: " << idx;
    ASSERT_EQ(e_0.index, 0);  // feature 0

    auto e_1 = data.at(idx+1);
    ASSERT_NEAR(e_1.fvalue, i * 2.0, kRtEps);
    ASSERT_EQ(e_1.index, 1);  // feature 1
  }
  ASSERT_EQ(offset.back(), n_rows * n_cols);
  for (size_t i = 0; i < n_rows + 1; ++i) {
    ASSERT_EQ(offset[i], i * n_cols);
  }
  ASSERT_EQ(source->info.num_row_, n_rows);
  ASSERT_EQ(source->info.num_col_, n_cols);
}

TEST(SimpleCSRSource, FromColumnarDense) {
  constexpr size_t kRows {16};
  constexpr size_t kCols {2};
  std::vector<Json> columns;
  thrust::device_vector<float> d_data_0(kRows);
  thrust::device_vector<int32_t> d_data_1(kRows);
  columns.emplace_back(GenerateDenseColumn<float>("<f4", kRows, &d_data_0));
  columns.emplace_back(GenerateDenseColumn<int32_t>("<i4", kRows, &d_data_1));

  Json column_arr {columns};

  std::stringstream ss;
  Json::Dump(column_arr, &ss);
  std::string str = ss.str();

  // no missing value
  {
    std::unique_ptr<data::SimpleCSRSource> source (new data::SimpleCSRSource());
    source->CopyFrom(str.c_str(), false);
    TestDenseColumn(source, kRows, kCols);
  }

  // with missing value specified
  {
    std::unique_ptr<data::SimpleCSRSource> source (new data::SimpleCSRSource());
    source->CopyFrom(str.c_str(), true, 4.0);

    auto const& data = source->page_.data.HostVector();
    auto const& offset = source->page_.offset.HostVector();
    ASSERT_EQ(data.size(), kRows * kCols - 2);
    ASSERT_NEAR(data[4].fvalue, 6.0, kRtEps);  // kCols * 2
    ASSERT_EQ(offset.back(), 30);
    for (size_t i = 3; i < kRows + 1; ++i) {
      ASSERT_EQ(offset[i], (i - 1) * 2);
    }
    ASSERT_EQ(source->info.num_row_, kRows);
    ASSERT_EQ(source->info.num_col_, kCols);
  }

  {
    // no missing value, but has NaN
    std::unique_ptr<data::SimpleCSRSource> source (new data::SimpleCSRSource());
    d_data_0[3] = std::numeric_limits<float>::quiet_NaN();
    ASSERT_TRUE(std::isnan(d_data_0[3]));  // removes 6.0
    source->CopyFrom(str.c_str(), false);

    auto const& data = source->page_.data.HostVector();
    auto const& offset = source->page_.offset.HostVector();
    ASSERT_EQ(data.size(), kRows * kCols - 1);
    ASSERT_NEAR(data[7].fvalue, 8.0, kRtEps);
    ASSERT_EQ(source->info.num_row_, kRows);
    ASSERT_EQ(source->info.num_col_, kCols);
  }
}

TEST(SimpleCSRSource, FromColumnarWithEmptyRows) {
  constexpr size_t kRows = 102;
  constexpr size_t kCols = 24;

  std::vector<Json> v_columns (kCols);
  std::vector<dh::device_vector<float>> columns_data(kCols);
  std::vector<dh::device_vector<RBitField8::value_type>> column_bitfields(kCols);

  RBitField8::value_type constexpr kUCOne = 1;

  for (size_t i = 0; i < kCols; ++i) {
    auto& col = v_columns[i];
    col = Object();
    auto& data = columns_data[i];
    data.resize(kRows);
    thrust::sequence(data.begin(), data.end(), 0);
    dh::safe_cuda(hipDeviceSynchronize());
    dh::safe_cuda(hipGetLastError());

    ASSERT_EQ(data.size(), kRows);

    auto p_d_data = raw_pointer_cast(data.data());
    std::vector<Json> j_data {
      Json(Integer(reinterpret_cast<Integer::Int>(p_d_data))),
          Json(Boolean(false))};
    col["data"] = j_data;
    std::vector<Json> j_shape {Json(Integer(static_cast<Integer::Int>(kRows)))};
    col["shape"] = Array(j_shape);
    col["version"] = Integer(static_cast<Integer::Int>(1));
    col["typestr"] = String("<f4");

    // Construct the mask object.
    col["mask"] = Object();
    auto& j_mask = col["mask"];
    j_mask["version"] = Integer(static_cast<Integer::Int>(1));
    auto& mask_storage = column_bitfields[i];
    mask_storage.resize(16);  // 16 bytes

    mask_storage[0] = ~(kUCOne << 2);  // 3^th row is missing
    mask_storage[1] = ~(kUCOne << 3);  // 12^th row is missing
    size_t last_ind = 12;
    mask_storage[last_ind] = ~(kUCOne << 5);
    std::set<size_t> missing_row_index {0, 1, last_ind};

    for (size_t i = 0; i < mask_storage.size(); ++i) {
      if (missing_row_index.find(i) == missing_row_index.cend()) {
        // all other rows are valid
        mask_storage[i] = ~0;
      }
    }

    j_mask["data"] = std::vector<Json>{
      Json(Integer(reinterpret_cast<Integer::Int>(mask_storage.data().get()))),
      Json(Boolean(false))};
    j_mask["shape"] = Array(std::vector<Json>{Json(Integer(static_cast<Integer::Int>(kRows)))});
    j_mask["typestr"] = String("|i1");
  }

  Json column_arr {Array(v_columns)};
  std::stringstream ss;
  Json::Dump(column_arr, &ss);
  std::string str = ss.str();
  std::unique_ptr<data::SimpleCSRSource> source (new data::SimpleCSRSource());
  source->CopyFrom(str.c_str(), false);

  auto const& data = source->page_.data.HostVector();
  auto const& offset = source->page_.offset.HostVector();

  ASSERT_EQ(offset.size(), kRows + 1);
  for (size_t i = 1; i < offset.size(); ++i) {
    for (size_t j = offset[i-1]; j < offset[i]; ++j) {
      ASSERT_EQ(data[j].index, j % kCols);
      ASSERT_NEAR(data[j].fvalue, i - 1, kRtEps);
    }
  }
  ASSERT_EQ(source->info.num_row_, kRows);
}

TEST(SimpleCSRSource, FromColumnarSparse) {
  constexpr size_t kRows = 32;
  constexpr size_t kCols = 2;
  RBitField8::value_type constexpr kUCOne = 1;

  std::vector<dh::device_vector<float>> columns_data(kCols);
  std::vector<dh::device_vector<RBitField8::value_type>> column_bitfields(kCols);

  {
    // column 0
    auto& mask = column_bitfields[0];
    mask.resize(8);

    for (size_t j = 0; j < mask.size(); ++j) {
      mask[j] = ~0;
    }
    // the 2^th entry of first column is invalid
    // [0 0 0 0 0 1 0 0]
    mask[0] = ~(kUCOne << 2);
  }
  {
    // column 1
    auto& mask = column_bitfields[1];
    mask.resize(8);

    for (size_t j = 0; j < mask.size(); ++j) {
      mask[j] = ~0;
    }
    // the 19^th entry of second column is invalid
    // [~0~], [~0~], [0 0 0 0 1 0 0 0]
    mask[2] = ~(kUCOne << 3);
  }

  for (size_t c = 0; c < kCols; ++c) {
    columns_data[c].resize(kRows);
    thrust::sequence(columns_data[c].begin(), columns_data[c].end(), 0);
  }

  std::vector<Json> j_columns(kCols);

  for (size_t c = 0; c < kCols; ++c) {
    auto& column = j_columns[c];
    column = Object();
    column["version"] = Integer(static_cast<Integer::Int>(1));
    column["typestr"] = String("<f4");
    auto p_d_data = raw_pointer_cast(columns_data[c].data());
    std::vector<Json> j_data {
      Json(Integer(reinterpret_cast<Integer::Int>(p_d_data))),
          Json(Boolean(false))};
    column["data"] = j_data;
    std::vector<Json> j_shape {Json(Integer(static_cast<Integer::Int>(kRows)))};
    column["shape"] = Array(j_shape);
    column["version"] = Integer(static_cast<Integer::Int>(1));
    column["typestr"] = String("<f4");

    column["mask"] = Object();
    auto& j_mask = column["mask"];
    j_mask["version"] = Integer(static_cast<Integer::Int>(1));
    j_mask["data"] = std::vector<Json>{
      Json(Integer(reinterpret_cast<Integer::Int>(column_bitfields[c].data().get()))),
      Json(Boolean(false))};
    j_mask["shape"] = Array(std::vector<Json>{Json(Integer(static_cast<Integer::Int>(kRows)))});
    j_mask["typestr"] = String("|i1");
  }

  Json column_arr {Array(j_columns)};

  std::stringstream ss;
  Json::Dump(column_arr, &ss);
  std::string str = ss.str();

  {
    std::unique_ptr<data::SimpleCSRSource> source (new data::SimpleCSRSource());
    source->CopyFrom(str.c_str(), false);

    auto const& data = source->page_.data.HostVector();
    auto const& offset = source->page_.offset.HostVector();

    ASSERT_EQ(offset.size(), kRows + 1);
    ASSERT_EQ(data[4].index, 1);
    ASSERT_EQ(data[4].fvalue, 2);
    ASSERT_EQ(data[37].index, 0);
    ASSERT_EQ(data[37].fvalue, 19);
  }

  {
    // with missing value
    std::unique_ptr<data::SimpleCSRSource> source (new data::SimpleCSRSource());
    source->CopyFrom(str.c_str(), true, /*missing=*/2.0);

    auto const& data = source->page_.data.HostVector();
    ASSERT_NE(data[4].fvalue, 2.0);
  }

  {
    // no missing value, but has NaN
    std::unique_ptr<data::SimpleCSRSource> source (new data::SimpleCSRSource());
    columns_data[0][4] = std::numeric_limits<float>::quiet_NaN();  // 0^th column 4^th row
    ASSERT_TRUE(std::isnan(columns_data[0][4]));
    source->CopyFrom(str.c_str(), false);

    auto const& data = source->page_.data.HostVector();
    auto const& offset = source->page_.offset.HostVector();
    // Two invalid entries and one NaN, in CSC
    // 0^th column: 0, 1, 4, 5, 6, ..., kRows
    // 1^th column: 0, 1, 2, 3, ..., 19, 21, ..., kRows
    // Turning it into CSR:
    // | 0, 0 | 1, 1 | 2 | 3, 3 | 4 | ...
    ASSERT_EQ(data.size(), kRows * kCols - 3);
    ASSERT_EQ(data[4].index, 1);  // from 1^th column
    ASSERT_EQ(data[5].fvalue, 3.0);
    ASSERT_EQ(data[7].index, 1);  // from 1^th column
    ASSERT_EQ(data[7].fvalue, 4.0);

    ASSERT_EQ(data[offset[2]].fvalue, 2.0);
    ASSERT_EQ(data[offset[4]].fvalue, 4.0);
  }

  {
    // with NaN as missing value
    // NaN is already set up by above test
    std::unique_ptr<data::SimpleCSRSource> source (new data::SimpleCSRSource());
    source->CopyFrom(str.c_str(), true,
                     /*missing=*/std::numeric_limits<float>::quiet_NaN());

    auto const& data = source->page_.data.HostVector();
    ASSERT_EQ(data.size(), kRows * kCols - 1);
    ASSERT_EQ(data[8].fvalue, 4.0);
  }
}

TEST(SimpleCSRSource, Types) {
  // Test with different types of different size
  constexpr size_t kRows {16};
  constexpr size_t kCols {2};
  std::vector<Json> columns;
  thrust::device_vector<double> d_data_0(kRows);
  thrust::device_vector<uint32_t> d_data_1(kRows);

  columns.emplace_back(GenerateDenseColumn<double>("<f8", kRows, &d_data_0));
  columns.emplace_back(GenerateDenseColumn<uint32_t>("<u4", kRows, &d_data_1));

  Json column_arr {columns};

  std::stringstream ss;
  Json::Dump(column_arr, &ss);
  std::string str = ss.str();

  std::unique_ptr<data::SimpleCSRSource> source (new data::SimpleCSRSource());
  source->CopyFrom(str.c_str(), false);
  TestDenseColumn(source, kRows, kCols);
}

}  // namespace xgboost
