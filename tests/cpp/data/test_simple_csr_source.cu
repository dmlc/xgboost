// Copyright (c) 2019 by Contributors
#include <gtest/gtest.h>
#include <xgboost/data.h>
#include <xgboost/json.h>
#include <thrust/device_vector.h>

#include <memory>
#include "../../../src/common/bitfield.h"
#include "../../../src/common/device_helpers.cuh"
#include "../../../src/data/simple_csr_source.h"

namespace xgboost {

TEST(SimpleCSRSource, FromColumnarDense) {
  constexpr size_t kRows = 16;
  Json column { Object() };
  std::vector<Json> j_shape {Json(Integer(static_cast<Integer::Int>(kRows)))};
  column["shape"] = Array(j_shape);
  column["strides"] = Array(std::vector<Json>{Json(Integer(static_cast<Integer::Int>(4)))});

  thrust::device_vector<float> d_data(kRows);
  for (size_t i = 0; i < d_data.size(); ++i) {
    d_data[i] = i * 2.0;
  }

  auto p_d_data = dh::Raw(d_data);

  std::vector<Json> j_data {
        Json(Integer(reinterpret_cast<Integer::Int>(p_d_data))),
        Json(Boolean(false))};
  column["data"] = j_data;

  column["version"] = Integer(static_cast<Integer::Int>(1));
  column["typestr"] = String("<f4");
  Json column_arr {Array{std::vector<Json>{column}}};

  std::stringstream ss;
  Json::Dump(column_arr, &ss);
  std::string str = ss.str();

  std::unique_ptr<data::SimpleCSRSource> source (new data::SimpleCSRSource());
  source->CopyFrom(str.c_str());

  auto const& data = source->page_.data.HostVector();
  auto const& offset = source->page_.offset.HostVector();
  for (size_t i = 0; i < kRows; ++i) {
    auto e = data[i];
    ASSERT_NEAR(e.fvalue, i * 2.0, kRtEps);
    ASSERT_EQ(e.index, 0);  // feature 0
  }
  ASSERT_EQ(offset.back(), 16);
  for (size_t i = 0; i < kRows + 1; ++i) {
    ASSERT_EQ(offset[i], i);
  }
}

TEST(SimpleCSRSource, FromColumnarWithEmptyRows) {
  // In this test we construct a data storage similar to cudf
  constexpr size_t kRows = 102;
  constexpr size_t kCols = 24;
  constexpr size_t kMissingRows = 3;

  std::vector<Json> v_columns (kCols);
  std::vector<dh::device_vector<float>> columns_data(kCols);
  std::vector<dh::device_vector<unsigned char>> column_bitfields(kCols);

  unsigned char constexpr kUCOne = 1;

  for (size_t i = 0; i < kCols; ++i) {
    auto& col = v_columns[i];
    col = Object();
    auto& data = columns_data[i];
    data.resize(kRows);
    thrust::sequence(data.begin(), data.end(), 0);
    dh::safe_cuda(hipDeviceSynchronize());
    dh::safe_cuda(hipGetLastError());

    ASSERT_EQ(data.size(), kRows);

    auto p_d_data = raw_pointer_cast(data.data());
    std::vector<Json> j_data {
      Json(Integer(reinterpret_cast<Integer::Int>(p_d_data))),
          Json(Boolean(false))};
    col["data"] = j_data;
    std::vector<Json> j_shape {Json(Integer(static_cast<Integer::Int>(kRows)))};
    col["shape"] = Array(j_shape);
    col["version"] = Integer(static_cast<Integer::Int>(1));
    col["typestr"] = String("<f4");

    // Construct the mask object.
    col["mask"] = Object();
    auto& j_mask = col["mask"];
    auto& mask_storage = column_bitfields[i];
    mask_storage.resize(16);  // 16 bytes

    mask_storage[0] = ~(kUCOne << 2);  // 3^th row is missing
    mask_storage[1] = ~(kUCOne << 3);  // 12^th row is missing
    size_t last_ind = 12;
    mask_storage[last_ind] = ~(kUCOne << 5);
    std::set<size_t> missing_row_index {0, 1, last_ind};

    for (size_t i = 0; i < mask_storage.size(); ++i) {
      if (missing_row_index.find(i) == missing_row_index.cend()) {
        // all other rows are valid
        mask_storage[i] = ~0;
      }
    }

    j_mask["data"] = std::vector<Json>{
      Json(Integer(reinterpret_cast<Integer::Int>(mask_storage.data().get()))),
      Json(Boolean(false))};
    j_mask["shape"] = Array(std::vector<Json>{Json(Integer(static_cast<Integer::Int>(16)))});
    j_mask["typestr"] = String("|i1");
    j_mask["null_count"] = Json(Integer(static_cast<Integer::Int>(kMissingRows)));
  }

  Json column_arr {Array(v_columns)};
  std::stringstream ss;
  Json::Dump(column_arr, &ss);
  std::string str = ss.str();
  std::unique_ptr<data::SimpleCSRSource> source (new data::SimpleCSRSource());
  source->CopyFrom(str.c_str());

  auto const& data = source->page_.data.HostVector();
  auto const& offset = source->page_.offset.HostVector();

  ASSERT_EQ(offset.size(), kRows + 1);
  for (size_t i = 1; i < offset.size(); ++i) {
    for (size_t j = offset[i-1]; j < offset[i]; ++j) {
      ASSERT_EQ(data[j].index, j % kCols);
      ASSERT_NEAR(data[j].fvalue, i - 1, kRtEps);
    }
  }
}

TEST(SimpleCSRSource, FromColumnarSparse) {
  constexpr size_t kRows = 32;
  constexpr size_t kCols = 2;
  unsigned char constexpr kUCOne = 1;

  std::vector<dh::device_vector<float>> columns_data(kCols);
  std::vector<dh::device_vector<unsigned char>> column_bitfields(kCols);

  {
    // column 0
    auto& mask = column_bitfields[0];
    mask.resize(8);

    for (size_t j = 0; j < mask.size(); ++j) {
      mask[j] = ~0;
    }
    mask[0] = ~(kUCOne << 2);
  }
  {
    // column 1
    auto& mask = column_bitfields[1];
    mask.resize(8);

    for (size_t j = 0; j < mask.size(); ++j) {
      mask[j] = ~0;
    }
    mask[2] = ~(kUCOne << 3);
  }

  for (size_t c = 0; c < kCols; ++c) {
    columns_data[c].resize(kRows);
    thrust::sequence(columns_data[c].begin(), columns_data[c].end(), 0);
  }

  std::vector<Json> j_columns(kCols);

  for (size_t c = 0; c < kCols; ++c) {
    auto& column = j_columns[c];
    column = Object();
    column["version"] = Integer(static_cast<Integer::Int>(1));
    column["typestr"] = String("<f4");
    auto p_d_data = raw_pointer_cast(columns_data[c].data());
    std::vector<Json> j_data {
      Json(Integer(reinterpret_cast<Integer::Int>(p_d_data))),
          Json(Boolean(false))};
    column["data"] = j_data;
    std::vector<Json> j_shape {Json(Integer(static_cast<Integer::Int>(kRows)))};
    column["shape"] = Array(j_shape);
    column["version"] = Integer(static_cast<Integer::Int>(1));
    column["typestr"] = String("<f4");

    column["mask"] = Object();
    auto& j_mask = column["mask"];
    j_mask["data"] = std::vector<Json>{
      Json(Integer(reinterpret_cast<Integer::Int>(column_bitfields[c].data().get()))),
      Json(Boolean(false))};
    j_mask["shape"] = Array(std::vector<Json>{Json(Integer(static_cast<Integer::Int>(8)))});
    j_mask["typestr"] = String("|i1");
    j_mask["null_count"] = Json(Integer(static_cast<Integer::Int>(1)));
  }

  Json column_arr {Array(j_columns)};

  std::stringstream ss;
  Json::Dump(column_arr, &ss);
  std::string str = ss.str();

  std::unique_ptr<data::SimpleCSRSource> source (new data::SimpleCSRSource());
  source->CopyFrom(str.c_str());

  auto const& data = source->page_.data.HostVector();
  auto const& offset = source->page_.offset.HostVector();

  ASSERT_EQ(offset.size(), kRows + 1);
  ASSERT_EQ(data[4].index, 1);
  ASSERT_EQ(data[4].fvalue, 2);
  ASSERT_EQ(data[37].index, 0);
  ASSERT_EQ(data[37].fvalue, 19);
}

}  // namespace xgboost