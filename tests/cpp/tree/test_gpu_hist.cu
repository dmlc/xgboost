/*!
 * Copyright 2017-2020 XGBoost contributors
 */
#include <gtest/gtest.h>
#include <thrust/device_vector.h>
#include <dmlc/filesystem.h>
#include <xgboost/base.h>
#include <random>
#include <string>
#include <vector>

#include "../helpers.h"
#include "../histogram_helpers.h"

#include "xgboost/json.h"
#include "../../../src/data/sparse_page_source.h"
#include "../../../src/tree/updater_gpu_hist.cu"
#include "../../../src/tree/updater_gpu_common.cuh"
#include "../../../src/common/common.h"
#include "../../../src/tree/constraints.cuh"

namespace xgboost {
namespace tree {

TEST(GpuHist, DeviceHistogram) {
  // Ensures that node allocates correctly after reaching `kStopGrowingSize`.
  dh::safe_cuda(hipSetDevice(0));
  constexpr size_t kNBins = 128;
  constexpr size_t kNNodes = 4;
  constexpr size_t kStopGrowing = kNNodes * kNBins * 2u;
  DeviceHistogram<GradientPairPrecise, kStopGrowing> histogram;
  histogram.Init(0, kNBins);
  for (size_t i = 0; i < kNNodes; ++i) {
    histogram.AllocateHistogram(i);
  }
  histogram.Reset();
  ASSERT_EQ(histogram.Data().size(), kStopGrowing);

  // Use allocated memory but do not erase nidx_map.
  for (size_t i = 0; i < kNNodes; ++i) {
    histogram.AllocateHistogram(i);
  }
  for (size_t i = 0; i < kNNodes; ++i) {
    ASSERT_TRUE(histogram.HistogramExists(i));
  }

  // Erase existing nidx_map.
  for (size_t i = kNNodes; i < kNNodes * 2; ++i) {
    histogram.AllocateHistogram(i);
  }
  for (size_t i = 0; i < kNNodes; ++i) {
    ASSERT_FALSE(histogram.HistogramExists(i));
  }
}

std::vector<GradientPairPrecise> GetHostHistGpair() {
  // 24 bins, 3 bins for each feature (column).
  std::vector<GradientPairPrecise> hist_gpair = {
    {0.8314f, 0.7147f}, {1.7989f, 3.7312f}, {3.3846f, 3.4598f},
    {2.9277f, 3.5886f}, {1.8429f, 2.4152f}, {1.2443f, 1.9019f},
    {1.6380f, 2.9174f}, {1.5657f, 2.5107f}, {2.8111f, 2.4776f},
    {2.1322f, 3.0651f}, {3.2927f, 3.8540f}, {0.5899f, 0.9866f},
    {1.5185f, 1.6263f}, {2.0686f, 3.1844f}, {2.4278f, 3.0950f},
    {1.5105f, 2.1403f}, {2.6922f, 4.2217f}, {1.8122f, 1.5437f},
    {0.0000f, 0.0000f}, {4.3245f, 5.7955f}, {1.6903f, 2.1103f},
    {2.4012f, 4.4754f}, {3.6136f, 3.4303f}, {0.0000f, 0.0000f}
  };
  return hist_gpair;
}

template <typename GradientSumT>
void TestBuildHist(bool use_shared_memory_histograms) {
  int const kNRows = 16, kNCols = 8;

  TrainParam param;
  std::vector<std::pair<std::string, std::string>> args {
    {"max_depth", "6"},
    {"max_leaves", "0"},
  };
  param.Init(args);
  auto page = BuildEllpackPage(kNRows, kNCols);
  BatchParam batch_param{};
  GPUHistMakerDevice<GradientSumT> maker(0, page.get(), {}, kNRows, param, kNCols, kNCols,
                                         true, batch_param);
  xgboost::SimpleLCG gen;
  xgboost::SimpleRealUniformDistribution<bst_float> dist(0.0f, 1.0f);
  HostDeviceVector<GradientPair> gpair(kNRows);
  for (auto &gp : gpair.HostVector()) {
    bst_float grad = dist(&gen);
    bst_float hess = dist(&gen);
    gp = GradientPair(grad, hess);
  }
  gpair.SetDevice(0);

  thrust::host_vector<common::CompressedByteT> h_gidx_buffer (page->gidx_buffer.HostVector());


  maker.row_partitioner.reset(new RowPartitioner(0, kNRows));
  maker.hist.AllocateHistogram(0);
  maker.gpair = gpair.DeviceSpan();

  maker.BuildHist(0);
  DeviceHistogram<GradientSumT> d_hist = maker.hist;

  auto node_histogram = d_hist.GetNodeHistogram(0);
  // d_hist.data stored in float, not gradient pair
  thrust::host_vector<GradientSumT> h_result (d_hist.Data().size() / 2);
  size_t data_size =
      sizeof(GradientSumT) /
      (sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT));
  data_size *= d_hist.Data().size();
  dh::safe_cuda(hipMemcpy(h_result.data(), node_histogram.data(), data_size,
                           hipMemcpyDeviceToHost));

  std::vector<GradientPairPrecise> solution = GetHostHistGpair();
  std::cout << std::fixed;
  for (size_t i = 0; i < h_result.size(); ++i) {
    EXPECT_NEAR(h_result[i].GetGrad(), solution[i].GetGrad(), 0.01f);
    EXPECT_NEAR(h_result[i].GetHess(), solution[i].GetHess(), 0.01f);
  }
}

TEST(GpuHist, BuildHistGlobalMem) {
  TestBuildHist<GradientPairPrecise>(false);
  TestBuildHist<GradientPair>(false);
}

TEST(GpuHist, BuildHistSharedMem) {
  TestBuildHist<GradientPairPrecise>(true);
  TestBuildHist<GradientPair>(true);
}

TEST(GpuHist, ApplySplit) {
  RegTree tree;
  ExpandEntry candidate;
  candidate.nid = 0;
  candidate.left_weight = 1.0f;
  candidate.right_weight = 2.0f;
  candidate.base_weight = 3.0f;
  candidate.split.is_cat = true;
  candidate.split.fvalue = 1.0f;  // at cat 1

  size_t n_rows = 10;
  size_t n_cols = 10;

  auto m = RandomDataGenerator{n_rows, n_cols, 0}.GenerateDMatrix(true);
  GenericParameter p;
  p.InitAllowUnknown(Args{});

  TrainParam tparam;
  tparam.InitAllowUnknown(Args{});
  BatchParam bparam;
  bparam.gpu_id = 0;
  bparam.max_bin = 3;
  bparam.gpu_page_size = 0;

  for (auto& ellpack : m->GetBatches<EllpackPage>(bparam)){
    auto impl = ellpack.Impl();
    HostDeviceVector<FeatureType> feature_types(10, FeatureType::kCategorical);
    feature_types.SetDevice(bparam.gpu_id);
    tree::GPUHistMakerDevice<GradientPairPrecise> updater(
        0, impl, feature_types.ConstDeviceSpan(), n_rows, tparam, 0, n_cols, true, bparam);
    updater.ApplySplit(candidate, &tree);

    ASSERT_EQ(tree.GetSplitTypes().size(), 3);
    ASSERT_EQ(tree.GetSplitTypes()[0], FeatureType::kCategorical);
    ASSERT_EQ(tree.GetSplitCategories().size(), 1);
    uint32_t bits = 1u << 30;  // bits: 0, 1, 0, 0, 0, ..., 0
    ASSERT_EQ(tree.GetSplitCategories().back(), bits);

    ASSERT_EQ(updater.node_categories.size(), 1);
  }
}

HistogramCutsWrapper GetHostCutMatrix () {
  HistogramCutsWrapper cmat;
  cmat.SetPtrs({0, 3, 6, 9, 12, 15, 18, 21, 24});
  cmat.SetMins({0.1f, 0.2f, 0.3f, 0.1f, 0.2f, 0.3f, 0.2f, 0.2f});
  // 24 cut fields, 3 cut fields for each feature (column).
  // Each row of the cut represents the cuts for a data column.
  cmat.SetValues({0.30f, 0.67f, 1.64f,
              0.32f, 0.77f, 1.95f,
              0.29f, 0.70f, 1.80f,
              0.32f, 0.75f, 1.85f,
              0.18f, 0.59f, 1.69f,
              0.25f, 0.74f, 2.00f,
              0.26f, 0.74f, 1.98f,
              0.26f, 0.71f, 1.83f});
  return cmat;
}

// TODO(trivialfis): This test is over simplified.
TEST(GpuHist, EvaluateRootSplit) {
  constexpr int kNRows = 16;
  constexpr int kNCols = 8;

  TrainParam param;

  std::vector<std::pair<std::string, std::string>> args{
      {"max_depth", "1"},
      {"max_leaves", "0"},

      // Disable all other parameters.
      {"colsample_bynode", "1"},
      {"colsample_bylevel", "1"},
      {"colsample_bytree", "1"},
      {"min_child_weight", "0.01"},
      {"reg_alpha", "0"},
      {"reg_lambda", "0"},
      {"max_delta_step", "0"}};
  param.Init(args);
  for (size_t i = 0; i < kNCols; ++i) {
    param.monotone_constraints.emplace_back(0);
  }

  int max_bins = 4;

  // Initialize GPUHistMakerDevice
  auto page = BuildEllpackPage(kNRows, kNCols);
  BatchParam batch_param{};
  GPUHistMakerDevice<GradientPairPrecise>
      maker(0, page.get(), {}, kNRows, param, kNCols, kNCols, true, batch_param);
  // Initialize GPUHistMakerDevice::node_sum_gradients
  maker.node_sum_gradients = {};

  // Initialize GPUHistMakerDevice::cut
  auto cmat = GetHostCutMatrix();

  // Copy cut matrix to device.
  page->Cuts() = cmat;
  maker.monotone_constraints = param.monotone_constraints;

  // Initialize GPUHistMakerDevice::hist
  maker.hist.Init(0, (max_bins - 1) * kNCols);
  maker.hist.AllocateHistogram(0);
  // Each row of hist_gpair represents gpairs for one feature.
  // Each entry represents a bin.
  std::vector<GradientPairPrecise> hist_gpair = GetHostHistGpair();
  std::vector<bst_float> hist;
  for (auto pair : hist_gpair) {
    hist.push_back(pair.GetGrad());
    hist.push_back(pair.GetHess());
  }

  ASSERT_EQ(maker.hist.Data().size(), hist.size());
  thrust::copy(hist.begin(), hist.end(),
    maker.hist.Data().begin());
  std::vector<float> feature_weights;

  maker.column_sampler.Init(kNCols, feature_weights, param.colsample_bynode,
                            param.colsample_bylevel, param.colsample_bytree,
                            false);

  RegTree tree;
  MetaInfo info;
  info.num_row_ = kNRows;
  info.num_col_ = kNCols;

  DeviceSplitCandidate res = maker.EvaluateRootSplit({6.4f, 12.8f});

  ASSERT_EQ(res.findex, 7);
  ASSERT_NEAR(res.fvalue, 0.26, xgboost::kRtEps);
}

void TestHistogramIndexImpl() {
  // Test if the compressed histogram index matches when using a sparse
  // dmatrix with and without using external memory

  int constexpr kNRows = 1000, kNCols = 10;

  // Build 2 matrices and build a histogram maker with that
  tree::GPUHistMakerSpecialised<GradientPairPrecise> hist_maker, hist_maker_ext;
  std::unique_ptr<DMatrix> hist_maker_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 0, true));

  dmlc::TemporaryDirectory tempdir;
  std::unique_ptr<DMatrix> hist_maker_ext_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 128UL, true, tempdir));

  std::vector<std::pair<std::string, std::string>> training_params = {
    {"max_depth", "10"},
    {"max_leaves", "0"}
  };

  GenericParameter generic_param(CreateEmptyGenericParam(0));
  hist_maker.Configure(training_params, &generic_param);
  hist_maker.InitDataOnce(hist_maker_dmat.get());
  hist_maker_ext.Configure(training_params, &generic_param);
  hist_maker_ext.InitDataOnce(hist_maker_ext_dmat.get());

  // Extract the device maker from the histogram makers and from that its compressed
  // histogram index
  const auto &maker = hist_maker.maker;
  std::vector<common::CompressedByteT> h_gidx_buffer(maker->page->gidx_buffer.HostVector());

  const auto &maker_ext = hist_maker_ext.maker;
  std::vector<common::CompressedByteT> h_gidx_buffer_ext(maker_ext->page->gidx_buffer.HostVector());

  ASSERT_EQ(maker->page->Cuts().TotalBins(), maker_ext->page->Cuts().TotalBins());
  ASSERT_EQ(maker->page->gidx_buffer.Size(), maker_ext->page->gidx_buffer.Size());
}

TEST(GpuHist, TestHistogramIndex) {
  TestHistogramIndexImpl();
}

// gamma is an alias of min_split_loss
int32_t TestMinSplitLoss(DMatrix* dmat, float gamma, HostDeviceVector<GradientPair>* gpair) {
  Args args {
    {"max_depth", "1"},
    {"max_leaves", "0"},

    // Disable all other parameters.
    {"colsample_bynode", "1"},
    {"colsample_bylevel", "1"},
    {"colsample_bytree", "1"},
    {"min_child_weight", "0.01"},
    {"reg_alpha", "0"},
    {"reg_lambda", "0"},
    {"max_delta_step", "0"},

    // test gamma
    {"gamma", std::to_string(gamma)}
  };

  tree::GPUHistMakerSpecialised<GradientPairPrecise> hist_maker;
  GenericParameter generic_param(CreateEmptyGenericParam(0));
  hist_maker.Configure(args, &generic_param);

  RegTree tree;
  hist_maker.Update(gpair, dmat, {&tree});

  auto n_nodes = tree.NumExtraNodes();
  return n_nodes;
}

TEST(GpuHist, MinSplitLoss) {
  constexpr size_t kRows = 32;
  constexpr size_t kCols = 16;
  constexpr float kSparsity = 0.6;
  auto dmat = RandomDataGenerator(kRows, kCols, kSparsity).Seed(3).GenerateDMatrix();
  auto gpair = GenerateRandomGradients(kRows);

  {
    int32_t n_nodes = TestMinSplitLoss(dmat.get(), 0.01, &gpair);
    // This is not strictly verified, meaning the numeber `2` is whatever GPU_Hist retured
    // when writing this test, and only used for testing larger gamma (below) does prevent
    // building tree.
    ASSERT_EQ(n_nodes, 2);
  }
  {
    int32_t n_nodes = TestMinSplitLoss(dmat.get(), 100.0, &gpair);
    // No new nodes with gamma == 100.
    ASSERT_EQ(n_nodes, static_cast<decltype(n_nodes)>(0));
  }
}

void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
                size_t gpu_page_size, RegTree* tree,
                HostDeviceVector<bst_float>* preds, float subsample = 1.0f,
                const std::string& sampling_method = "uniform",
                int max_bin = 2) {

  if (gpu_page_size > 0) {
    // Loop over the batches and count the records
    int64_t batch_count = 0;
    int64_t row_count = 0;
    for (const auto& batch : dmat->GetBatches<EllpackPage>({0, max_bin, gpu_page_size})) {
      EXPECT_LT(batch.Size(), dmat->Info().num_row_);
      batch_count++;
      row_count += batch.Size();
    }
    EXPECT_GE(batch_count, 2);
    EXPECT_EQ(row_count, dmat->Info().num_row_);
  }

  Args args{
      {"max_depth", "2"},
      {"max_bin", std::to_string(max_bin)},
      {"min_child_weight", "0.0"},
      {"reg_alpha", "0"},
      {"reg_lambda", "0"},
      {"subsample", std::to_string(subsample)},
      {"sampling_method", sampling_method},
  };

  tree::GPUHistMakerSpecialised<GradientPairPrecise> hist_maker;
  GenericParameter generic_param(CreateEmptyGenericParam(0));
  generic_param.gpu_page_size = gpu_page_size;
  hist_maker.Configure(args, &generic_param);

  hist_maker.Update(gpair, dmat, {tree});
  hist_maker.UpdatePredictionCache(dmat, preds);
}

TEST(GpuHist, UniformSampling) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr float kSubsample = 0.9999;
  common::GlobalRandom().seed(1994);

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  auto gpair = GenerateRandomGradients(kRows);

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree, &preds, 1.0, "uniform", kRows);
  // Build another tree using sampling.
  RegTree tree_sampling;
  HostDeviceVector<bst_float> preds_sampling(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree_sampling, &preds_sampling, kSubsample,
             "uniform", kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_sampling_h = preds_sampling.ConstHostVector();
  for (int i = 0; i < kRows; i++) {
    EXPECT_NEAR(preds_h[i], preds_sampling_h[i], 1e-8);
  }
}

TEST(GpuHist, GradientBasedSampling) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr float kSubsample = 0.9999;
  common::GlobalRandom().seed(1994);

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  auto gpair = GenerateRandomGradients(kRows);

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree, &preds, 1.0, "uniform", kRows);

  // Build another tree using sampling.
  RegTree tree_sampling;
  HostDeviceVector<bst_float> preds_sampling(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree_sampling, &preds_sampling, kSubsample,
             "gradient_based", kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_sampling_h = preds_sampling.ConstHostVector();
  for (int i = 0; i < kRows; i++) {
    EXPECT_NEAR(preds_h[i], preds_sampling_h[i], 1e-3);
  }
}

TEST(GpuHist, ExternalMemory) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr size_t kPageSize = 1024;

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  // Create a DMatrix with multiple batches.
  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix>
      dmat_ext(CreateSparsePageDMatrixWithRC(kRows, kCols, kPageSize, true, tmpdir));

  auto gpair = GenerateRandomGradients(kRows);

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree, &preds, 1.0, "uniform", kRows);
  // Build another tree using multiple ELLPACK pages.
  RegTree tree_ext;
  HostDeviceVector<bst_float> preds_ext(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat_ext.get(), kPageSize, &tree_ext, &preds_ext, 1.0, "uniform", kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_ext_h = preds_ext.ConstHostVector();
  for (int i = 0; i < kRows; i++) {
    EXPECT_NEAR(preds_h[i], preds_ext_h[i], 1e-6);
  }
}

TEST(GpuHist, ExternalMemoryWithSampling) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr size_t kPageSize = 1024;
  constexpr float kSubsample = 0.5;
  const std::string kSamplingMethod = "gradient_based";
  common::GlobalRandom().seed(0);

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  // Create a DMatrix with multiple batches.
  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix>
      dmat_ext(CreateSparsePageDMatrixWithRC(kRows, kCols, kPageSize, true, tmpdir));

  auto gpair = GenerateRandomGradients(kRows);

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree, &preds, kSubsample, kSamplingMethod,
             kRows);

  // Build another tree using multiple ELLPACK pages.
  RegTree tree_ext;
  HostDeviceVector<bst_float> preds_ext(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat_ext.get(), kPageSize, &tree_ext, &preds_ext,
             kSubsample, kSamplingMethod, kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_ext_h = preds_ext.ConstHostVector();
  for (int i = 0; i < kRows; i++) {
    EXPECT_NEAR(preds_h[i], preds_ext_h[i], 2e-3);
  }
}

TEST(GpuHist, ConfigIO) {
  GenericParameter generic_param(CreateEmptyGenericParam(0));
  std::unique_ptr<TreeUpdater> updater {TreeUpdater::Create("grow_gpu_hist", &generic_param) };
  updater->Configure(Args{});

  Json j_updater { Object() };
  updater->SaveConfig(&j_updater);
  ASSERT_TRUE(IsA<Object>(j_updater["gpu_hist_train_param"]));
  ASSERT_TRUE(IsA<Object>(j_updater["train_param"]));
  updater->LoadConfig(j_updater);

  Json j_updater_roundtrip { Object() };
  updater->SaveConfig(&j_updater_roundtrip);
  ASSERT_TRUE(IsA<Object>(j_updater_roundtrip["gpu_hist_train_param"]));
  ASSERT_TRUE(IsA<Object>(j_updater_roundtrip["train_param"]));

  ASSERT_EQ(j_updater, j_updater_roundtrip);
}

TEST(GpuHist, MaxDepth) {
  GenericParameter generic_param(CreateEmptyGenericParam(0));
  size_t constexpr kRows = 16;
  size_t constexpr kCols = 4;
  auto p_mat = RandomDataGenerator{kRows, kCols, 0}.GenerateDMatrix();

  auto learner = std::unique_ptr<Learner>(Learner::Create({p_mat}));
  learner->SetParam("max_depth", "32");
  learner->Configure();

  ASSERT_THROW({learner->UpdateOneIter(0, p_mat);}, dmlc::Error);
}
}  // namespace tree
}  // namespace xgboost
