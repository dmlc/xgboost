#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2022 XGBoost contributors
 */
#include <gtest/gtest.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <xgboost/base.h>

#include <random>
#include <string>
#include <vector>

#include "../../../src/common/common.h"
#include "../../../src/data/sparse_page_source.h"
#include "../../../src/tree/constraints.cuh"
#include "../../../src/tree/updater_gpu_common.cuh"
#include "../../../src/tree/updater_gpu_hist.cu"
#include "../filesystem.h"  // dmlc::TemporaryDirectory
#include "../helpers.h"
#include "../histogram_helpers.h"
#include "xgboost/generic_parameters.h"
#include "xgboost/json.h"

namespace xgboost {
namespace tree {
TEST(GpuHist, DeviceHistogram) {
  // Ensures that node allocates correctly after reaching `kStopGrowingSize`.
  dh::safe_cuda(hipSetDevice(0));
  constexpr size_t kNBins = 128;
  constexpr int kNNodes = 4;
  constexpr size_t kStopGrowing = kNNodes * kNBins * 2u;
  DeviceHistogramStorage<GradientPairPrecise, kStopGrowing> histogram;
  histogram.Init(0, kNBins);
  for (int i = 0; i < kNNodes; ++i) {
    histogram.AllocateHistograms({i});
  }
  histogram.Reset();
  ASSERT_EQ(histogram.Data().size(), kStopGrowing);

  // Use allocated memory but do not erase nidx_map.
  for (int i = 0; i < kNNodes; ++i) {
    histogram.AllocateHistograms({i});
  }
  for (int i = 0; i < kNNodes; ++i) {
    ASSERT_TRUE(histogram.HistogramExists(i));
  }

  // Add two new nodes
  histogram.AllocateHistograms({kNNodes});
  histogram.AllocateHistograms({kNNodes + 1});

  // Old cached nodes should still exist
  for (int i = 0; i < kNNodes; ++i) {
    ASSERT_TRUE(histogram.HistogramExists(i));
  }

  // Should be deleted
  ASSERT_FALSE(histogram.HistogramExists(kNNodes));
  // Most recent node should exist
  ASSERT_TRUE(histogram.HistogramExists(kNNodes + 1));

  // Add same node again - should fail
  EXPECT_ANY_THROW(histogram.AllocateHistograms({kNNodes + 1}););
}

std::vector<GradientPairPrecise> GetHostHistGpair() {
  // 24 bins, 3 bins for each feature (column).
  std::vector<GradientPairPrecise> hist_gpair = {
    {0.8314f, 0.7147f}, {1.7989f, 3.7312f}, {3.3846f, 3.4598f},
    {2.9277f, 3.5886f}, {1.8429f, 2.4152f}, {1.2443f, 1.9019f},
    {1.6380f, 2.9174f}, {1.5657f, 2.5107f}, {2.8111f, 2.4776f},
    {2.1322f, 3.0651f}, {3.2927f, 3.8540f}, {0.5899f, 0.9866f},
    {1.5185f, 1.6263f}, {2.0686f, 3.1844f}, {2.4278f, 3.0950f},
    {1.5105f, 2.1403f}, {2.6922f, 4.2217f}, {1.8122f, 1.5437f},
    {0.0000f, 0.0000f}, {4.3245f, 5.7955f}, {1.6903f, 2.1103f},
    {2.4012f, 4.4754f}, {3.6136f, 3.4303f}, {0.0000f, 0.0000f}
  };
  return hist_gpair;
}

template <typename GradientSumT>
void TestBuildHist(bool use_shared_memory_histograms) {
  int const kNRows = 16, kNCols = 8;

  TrainParam param;
  std::vector<std::pair<std::string, std::string>> args {
    {"max_depth", "6"},
    {"max_leaves", "0"},
  };
  param.Init(args);
  auto page = BuildEllpackPage(kNRows, kNCols);
  BatchParam batch_param{};
  Context ctx{CreateEmptyGenericParam(0)};
  GPUHistMakerDevice<GradientSumT> maker(&ctx, page.get(), {}, kNRows, param, kNCols, kNCols,
                                         batch_param);
  xgboost::SimpleLCG gen;
  xgboost::SimpleRealUniformDistribution<bst_float> dist(0.0f, 1.0f);
  HostDeviceVector<GradientPair> gpair(kNRows);
  for (auto &gp : gpair.HostVector()) {
    bst_float grad = dist(&gen);
    bst_float hess = dist(&gen);
    gp = GradientPair(grad, hess);
  }
  gpair.SetDevice(0);

  thrust::host_vector<common::CompressedByteT> h_gidx_buffer (page->gidx_buffer.HostVector());
  maker.row_partitioner.reset(new RowPartitioner(0, kNRows));
  maker.hist.AllocateHistograms({0});
  maker.gpair = gpair.DeviceSpan();
  maker.histogram_rounding = CreateRoundingFactor<GradientSumT>(maker.gpair);

  BuildGradientHistogram(
      page->GetDeviceAccessor(0), maker.feature_groups->DeviceAccessor(0),
      gpair.DeviceSpan(), maker.row_partitioner->GetRows(0),
      maker.hist.GetNodeHistogram(0), maker.histogram_rounding,
      !use_shared_memory_histograms);

  DeviceHistogramStorage<GradientSumT>& d_hist = maker.hist;

  auto node_histogram = d_hist.GetNodeHistogram(0);
  // d_hist.data stored in float, not gradient pair
  thrust::host_vector<GradientSumT> h_result (d_hist.Data().size() / 2);
  size_t data_size =
      sizeof(GradientSumT) /
      (sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT));
  data_size *= d_hist.Data().size();
  dh::safe_cuda(hipMemcpy(h_result.data(), node_histogram.data(), data_size,
                           hipMemcpyDeviceToHost));

  std::vector<GradientPairPrecise> solution = GetHostHistGpair();
  std::cout << std::fixed;
  for (size_t i = 0; i < h_result.size(); ++i) {
    ASSERT_FALSE(std::isnan(h_result[i].GetGrad()));
    EXPECT_NEAR(h_result[i].GetGrad(), solution[i].GetGrad(), 0.01f);
    EXPECT_NEAR(h_result[i].GetHess(), solution[i].GetHess(), 0.01f);
  }
}

TEST(GpuHist, BuildHistGlobalMem) {
  TestBuildHist<GradientPairPrecise>(false);
}

TEST(GpuHist, BuildHistSharedMem) {
  TestBuildHist<GradientPairPrecise>(true);
}

HistogramCutsWrapper GetHostCutMatrix () {
  HistogramCutsWrapper cmat;
  cmat.SetPtrs({0, 3, 6, 9, 12, 15, 18, 21, 24});
  cmat.SetMins({0.1f, 0.2f, 0.3f, 0.1f, 0.2f, 0.3f, 0.2f, 0.2f});
  // 24 cut fields, 3 cut fields for each feature (column).
  // Each row of the cut represents the cuts for a data column.
  cmat.SetValues({0.30f, 0.67f, 1.64f,
              0.32f, 0.77f, 1.95f,
              0.29f, 0.70f, 1.80f,
              0.32f, 0.75f, 1.85f,
              0.18f, 0.59f, 1.69f,
              0.25f, 0.74f, 2.00f,
              0.26f, 0.74f, 1.98f,
              0.26f, 0.71f, 1.83f});
  return cmat;
}

// TODO(trivialfis): This test is over simplified.
TEST(GpuHist, EvaluateRootSplit) {
  constexpr int kNRows = 16;
  constexpr int kNCols = 8;

  TrainParam param;

  std::vector<std::pair<std::string, std::string>> args{
      {"max_depth", "1"},
      {"max_leaves", "0"},

      // Disable all other parameters.
      {"colsample_bynode", "1"},
      {"colsample_bylevel", "1"},
      {"colsample_bytree", "1"},
      {"min_child_weight", "0.01"},
      {"reg_alpha", "0"},
      {"reg_lambda", "0"},
      {"max_delta_step", "0"}};
  param.Init(args);
  for (size_t i = 0; i < kNCols; ++i) {
    param.monotone_constraints.emplace_back(0);
  }

  int max_bins = 4;

  // Initialize GPUHistMakerDevice
  auto page = BuildEllpackPage(kNRows, kNCols);
  BatchParam batch_param{};
  Context ctx{CreateEmptyGenericParam(0)};
  GPUHistMakerDevice<GradientPairPrecise> maker(&ctx, page.get(), {}, kNRows, param, kNCols, kNCols,
                                                batch_param);
  // Initialize GPUHistMakerDevice::node_sum_gradients
  maker.node_sum_gradients = {};

  // Initialize GPUHistMakerDevice::cut
  auto cmat = GetHostCutMatrix();

  // Copy cut matrix to device.
  page->Cuts() = cmat;
  maker.monotone_constraints = param.monotone_constraints;

  // Initialize GPUHistMakerDevice::hist
  maker.hist.Init(0, (max_bins - 1) * kNCols);
  maker.hist.AllocateHistograms({0});
  // Each row of hist_gpair represents gpairs for one feature.
  // Each entry represents a bin.
  std::vector<GradientPairPrecise> hist_gpair = GetHostHistGpair();
  std::vector<bst_float> hist;
  for (auto pair : hist_gpair) {
    hist.push_back(pair.GetGrad());
    hist.push_back(pair.GetHess());
  }

  ASSERT_EQ(maker.hist.Data().size(), hist.size());
  thrust::copy(hist.begin(), hist.end(),
    maker.hist.Data().begin());
  std::vector<float> feature_weights;

  maker.column_sampler.Init(kNCols, feature_weights, param.colsample_bynode,
                            param.colsample_bylevel, param.colsample_bytree);

  RegTree tree;
  MetaInfo info;
  info.num_row_ = kNRows;
  info.num_col_ = kNCols;

  DeviceSplitCandidate res = maker.EvaluateRootSplit({6.4f, 12.8f}).split;

  ASSERT_EQ(res.findex, 7);
  ASSERT_NEAR(res.fvalue, 0.26, xgboost::kRtEps);
}

void TestHistogramIndexImpl() {
  // Test if the compressed histogram index matches when using a sparse
  // dmatrix with and without using external memory

  int constexpr kNRows = 1000, kNCols = 10;

  // Build 2 matrices and build a histogram maker with that

  GenericParameter generic_param(CreateEmptyGenericParam(0));
  tree::GPUHistMaker hist_maker{&generic_param,ObjInfo{ObjInfo::kRegression}},
      hist_maker_ext{&generic_param,ObjInfo{ObjInfo::kRegression}};
  std::unique_ptr<DMatrix> hist_maker_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 0, true));

  dmlc::TemporaryDirectory tempdir;
  std::unique_ptr<DMatrix> hist_maker_ext_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 128UL, true, tempdir));

  std::vector<std::pair<std::string, std::string>> training_params = {
    {"max_depth", "10"},
    {"max_leaves", "0"}
  };

  hist_maker.Configure(training_params);
  hist_maker.InitDataOnce(hist_maker_dmat.get());
  hist_maker_ext.Configure(training_params);
  hist_maker_ext.InitDataOnce(hist_maker_ext_dmat.get());

  // Extract the device maker from the histogram makers and from that its compressed
  // histogram index
  const auto &maker = hist_maker.maker;
  auto grad = GenerateRandomGradients(kNRows);
  grad.SetDevice(0);
  maker->Reset(&grad, hist_maker_dmat.get(), kNCols);
  std::vector<common::CompressedByteT> h_gidx_buffer(maker->page->gidx_buffer.HostVector());

  const auto &maker_ext = hist_maker_ext.maker;
  maker_ext->Reset(&grad, hist_maker_ext_dmat.get(), kNCols);
  std::vector<common::CompressedByteT> h_gidx_buffer_ext(maker_ext->page->gidx_buffer.HostVector());

  ASSERT_EQ(maker->page->Cuts().TotalBins(), maker_ext->page->Cuts().TotalBins());
  ASSERT_EQ(maker->page->gidx_buffer.Size(), maker_ext->page->gidx_buffer.Size());
}

TEST(GpuHist, TestHistogramIndex) {
  TestHistogramIndexImpl();
}

void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
                size_t gpu_page_size, RegTree* tree,
                HostDeviceVector<bst_float>* preds, float subsample = 1.0f,
                const std::string& sampling_method = "uniform",
                int max_bin = 2) {

  if (gpu_page_size > 0) {
    // Loop over the batches and count the records
    int64_t batch_count = 0;
    int64_t row_count = 0;
    for (const auto& batch : dmat->GetBatches<EllpackPage>({0, max_bin})) {
      EXPECT_LT(batch.Size(), dmat->Info().num_row_);
      batch_count++;
      row_count += batch.Size();
    }
    EXPECT_GE(batch_count, 2);
    EXPECT_EQ(row_count, dmat->Info().num_row_);
  }

  Args args{
      {"max_depth", "2"},
      {"max_bin", std::to_string(max_bin)},
      {"min_child_weight", "0.0"},
      {"reg_alpha", "0"},
      {"reg_lambda", "0"},
      {"subsample", std::to_string(subsample)},
      {"sampling_method", sampling_method},
  };

  GenericParameter generic_param(CreateEmptyGenericParam(0));
  tree::GPUHistMaker hist_maker{&generic_param,ObjInfo{ObjInfo::kRegression}};
  hist_maker.Configure(args);

  std::vector<HostDeviceVector<bst_node_t>> position(1);
  hist_maker.Update(gpair, dmat, common::Span<HostDeviceVector<bst_node_t>>{position}, {tree});
  auto cache = linalg::VectorView<float>{preds->DeviceSpan(), {preds->Size()}, 0};
  hist_maker.UpdatePredictionCache(dmat, cache);
}

TEST(GpuHist, UniformSampling) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr float kSubsample = 0.9999;
  common::GlobalRandom().seed(1994);

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  auto gpair = GenerateRandomGradients(kRows);

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree, &preds, 1.0, "uniform", kRows);
  // Build another tree using sampling.
  RegTree tree_sampling;
  HostDeviceVector<bst_float> preds_sampling(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree_sampling, &preds_sampling, kSubsample,
             "uniform", kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_sampling_h = preds_sampling.ConstHostVector();
  for (size_t i = 0; i < kRows; i++) {
    EXPECT_NEAR(preds_h[i], preds_sampling_h[i], 1e-8);
  }
}

TEST(GpuHist, GradientBasedSampling) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr float kSubsample = 0.9999;
  common::GlobalRandom().seed(1994);

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  auto gpair = GenerateRandomGradients(kRows);

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree, &preds, 1.0, "uniform", kRows);

  // Build another tree using sampling.
  RegTree tree_sampling;
  HostDeviceVector<bst_float> preds_sampling(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree_sampling, &preds_sampling, kSubsample,
             "gradient_based", kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_sampling_h = preds_sampling.ConstHostVector();
  for (size_t i = 0; i < kRows; i++) {
    EXPECT_NEAR(preds_h[i], preds_sampling_h[i], 1e-3);
  }
}

TEST(GpuHist, ExternalMemory) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr size_t kPageSize = 1024;

  dmlc::TemporaryDirectory tmpdir;

  // Create a DMatrix with multiple batches.
  std::unique_ptr<DMatrix> dmat_ext(
      CreateSparsePageDMatrix(kRows, kCols, kRows / kPageSize, tmpdir.path + "/cache"));

  // Create a single batch DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrix(kRows, kCols, 1, tmpdir.path + "/cache"));

  auto gpair = GenerateRandomGradients(kRows);

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree, &preds, 1.0, "uniform", kRows);
  // Build another tree using multiple ELLPACK pages.
  RegTree tree_ext;
  HostDeviceVector<bst_float> preds_ext(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat_ext.get(), kPageSize, &tree_ext, &preds_ext, 1.0, "uniform", kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_ext_h = preds_ext.ConstHostVector();
  for (size_t i = 0; i < kRows; i++) {
    EXPECT_NEAR(preds_h[i], preds_ext_h[i], 1e-6);
  }
}

TEST(GpuHist, ExternalMemoryWithSampling) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr size_t kPageSize = 1024;
  constexpr float kSubsample = 0.5;
  const std::string kSamplingMethod = "gradient_based";
  common::GlobalRandom().seed(0);

  dmlc::TemporaryDirectory tmpdir;

  // Create a single batch DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrix(kRows, kCols, 1, tmpdir.path + "/cache"));

  // Create a DMatrix with multiple batches.
  std::unique_ptr<DMatrix> dmat_ext(
      CreateSparsePageDMatrix(kRows, kCols, kRows / kPageSize, tmpdir.path + "/cache"));

  auto gpair = GenerateRandomGradients(kRows);

  // Build a tree using the in-memory DMatrix.
  auto rng = common::GlobalRandom();

  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree, &preds, kSubsample, kSamplingMethod,
             kRows);

  // Build another tree using multiple ELLPACK pages.
  common::GlobalRandom() = rng;
  RegTree tree_ext;
  HostDeviceVector<bst_float> preds_ext(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat_ext.get(), kPageSize, &tree_ext, &preds_ext,
             kSubsample, kSamplingMethod, kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_ext_h = preds_ext.ConstHostVector();
  for (size_t i = 0; i < kRows; i++) {
    ASSERT_NEAR(preds_h[i], preds_ext_h[i], 1e-3);
  }
}

TEST(GpuHist, ConfigIO) {
  GenericParameter generic_param(CreateEmptyGenericParam(0));
  std::unique_ptr<TreeUpdater> updater{
      TreeUpdater::Create("grow_gpu_hist", &generic_param, ObjInfo{ObjInfo::kRegression})};
  updater->Configure(Args{});

  Json j_updater { Object() };
  updater->SaveConfig(&j_updater);
  ASSERT_TRUE(IsA<Object>(j_updater["gpu_hist_train_param"]));
  ASSERT_TRUE(IsA<Object>(j_updater["train_param"]));
  updater->LoadConfig(j_updater);

  Json j_updater_roundtrip { Object() };
  updater->SaveConfig(&j_updater_roundtrip);
  ASSERT_TRUE(IsA<Object>(j_updater_roundtrip["gpu_hist_train_param"]));
  ASSERT_TRUE(IsA<Object>(j_updater_roundtrip["train_param"]));

  ASSERT_EQ(j_updater, j_updater_roundtrip);
}

TEST(GpuHist, MaxDepth) {
  GenericParameter generic_param(CreateEmptyGenericParam(0));
  size_t constexpr kRows = 16;
  size_t constexpr kCols = 4;
  auto p_mat = RandomDataGenerator{kRows, kCols, 0}.GenerateDMatrix();

  auto learner = std::unique_ptr<Learner>(Learner::Create({p_mat}));
  learner->SetParam("max_depth", "32");
  learner->Configure();

  ASSERT_THROW({learner->UpdateOneIter(0, p_mat);}, dmlc::Error);
}
}  // namespace tree
}  // namespace xgboost
