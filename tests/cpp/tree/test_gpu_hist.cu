/*!
 * Copyright 2017-2019 XGBoost contributors
 */
#include <thrust/device_vector.h>
#include <dmlc/filesystem.h>
#include <xgboost/base.h>
#include <random>
#include <string>
#include <vector>

#include "../helpers.h"
#include "gtest/gtest.h"

#include "../../../src/data/sparse_page_source.h"
#include "../../../src/gbm/gbtree_model.h"
#include "../../../src/tree/updater_gpu_hist.cu"
#include "../../../src/tree/updater_gpu_common.cuh"
#include "../../../src/common/common.h"
#include "../../../src/tree/constraints.cuh"

namespace xgboost {
namespace tree {

TEST(GpuHist, DeviceHistogram) {
  // Ensures that node allocates correctly after reaching `kStopGrowingSize`.
  dh::SaveCudaContext{
    [&]() {
      dh::safe_cuda(hipSetDevice(0));
      constexpr size_t kNBins = 128;
      constexpr size_t kNNodes = 4;
      constexpr size_t kStopGrowing = kNNodes * kNBins * 2u;
      DeviceHistogram<GradientPairPrecise, kStopGrowing> histogram;
      histogram.Init(0, kNBins);
      for (size_t i = 0; i < kNNodes; ++i) {
        histogram.AllocateHistogram(i);
      }
      histogram.Reset();
      ASSERT_EQ(histogram.Data().size(), kStopGrowing);

      // Use allocated memory but do not erase nidx_map.
      for (size_t i = 0; i < kNNodes; ++i) {
        histogram.AllocateHistogram(i);
      }
      for (size_t i = 0; i < kNNodes; ++i) {
        ASSERT_TRUE(histogram.HistogramExists(i));
      }

      // Erase existing nidx_map.
      for (size_t i = kNNodes; i < kNNodes * 2; ++i) {
        histogram.AllocateHistogram(i);
      }
      for (size_t i = 0; i < kNNodes; ++i) {
        ASSERT_FALSE(histogram.HistogramExists(i));
      }
    }
  };
}

namespace {
class HistogramCutsWrapper : public common::HistogramCuts {
 public:
  using SuperT = common::HistogramCuts;
  void SetValues(std::vector<float> cuts) {
    SuperT::cut_values_ = cuts;
  }
  void SetPtrs(std::vector<uint32_t> ptrs) {
    SuperT::cut_ptrs_ = ptrs;
  }
  void SetMins(std::vector<float> mins) {
    SuperT::min_vals_ = mins;
  }
};
}  //  anonymous namespace

std::vector<GradientPairPrecise> GetHostHistGpair() {
  // 24 bins, 3 bins for each feature (column).
  std::vector<GradientPairPrecise> hist_gpair = {
    {0.8314f, 0.7147f}, {1.7989f, 3.7312f}, {3.3846f, 3.4598f},
    {2.9277f, 3.5886f}, {1.8429f, 2.4152f}, {1.2443f, 1.9019f},
    {1.6380f, 2.9174f}, {1.5657f, 2.5107f}, {2.8111f, 2.4776f},
    {2.1322f, 3.0651f}, {3.2927f, 3.8540f}, {0.5899f, 0.9866f},
    {1.5185f, 1.6263f}, {2.0686f, 3.1844f}, {2.4278f, 3.0950f},
    {1.5105f, 2.1403f}, {2.6922f, 4.2217f}, {1.8122f, 1.5437f},
    {0.0000f, 0.0000f}, {4.3245f, 5.7955f}, {1.6903f, 2.1103f},
    {2.4012f, 4.4754f}, {3.6136f, 3.4303f}, {0.0000f, 0.0000f}
  };
  return hist_gpair;
}

template <typename GradientSumT>
void TestBuildHist(bool use_shared_memory_histograms) {
  int const kNRows = 16, kNCols = 8;

  TrainParam param;
  std::vector<std::pair<std::string, std::string>> args {
    {"max_depth", "6"},
    {"max_leaves", "0"},
  };
  param.Init(args);
  auto page = BuildEllpackPage(kNRows, kNCols);
  GPUHistMakerDevice<GradientSumT> maker(0, page.get(), kNRows, param, kNCols, kNCols);
  maker.InitHistogram();

  xgboost::SimpleLCG gen;
  xgboost::SimpleRealUniformDistribution<bst_float> dist(0.0f, 1.0f);
  std::vector<GradientPair> h_gpair(kNRows);
  for (auto &gpair : h_gpair) {
    bst_float grad = dist(&gen);
    bst_float hess = dist(&gen);
    gpair = GradientPair(grad, hess);
  }

  thrust::host_vector<common::CompressedByteT> h_gidx_buffer (page->gidx_buffer.size());

  common::CompressedByteT* d_gidx_buffer_ptr = page->gidx_buffer.data();
  dh::safe_cuda(hipMemcpy(h_gidx_buffer.data(), d_gidx_buffer_ptr,
                           sizeof(common::CompressedByteT) * page->gidx_buffer.size(),
                           hipMemcpyDeviceToHost));

  maker.row_partitioner.reset(new RowPartitioner(0, kNRows));
  maker.hist.AllocateHistogram(0);
  dh::CopyVectorToDeviceSpan(maker.gpair, h_gpair);

  maker.use_shared_memory_histograms = use_shared_memory_histograms;
  maker.BuildHist(0);
  DeviceHistogram<GradientSumT> d_hist = maker.hist;

  auto node_histogram = d_hist.GetNodeHistogram(0);
  // d_hist.data stored in float, not gradient pair
  thrust::host_vector<GradientSumT> h_result (d_hist.Data().size() / 2);
  size_t data_size =
      sizeof(GradientSumT) /
      (sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT));
  data_size *= d_hist.Data().size();
  dh::safe_cuda(hipMemcpy(h_result.data(), node_histogram.data(), data_size,
                           hipMemcpyDeviceToHost));

  std::vector<GradientPairPrecise> solution = GetHostHistGpair();
  std::cout << std::fixed;
  for (size_t i = 0; i < h_result.size(); ++i) {
    EXPECT_NEAR(h_result[i].GetGrad(), solution[i].GetGrad(), 0.01f);
    EXPECT_NEAR(h_result[i].GetHess(), solution[i].GetHess(), 0.01f);
  }
}

TEST(GpuHist, BuildHistGlobalMem) {
  TestBuildHist<GradientPairPrecise>(false);
  TestBuildHist<GradientPair>(false);
}

TEST(GpuHist, BuildHistSharedMem) {
  TestBuildHist<GradientPairPrecise>(true);
  TestBuildHist<GradientPair>(true);
}

HistogramCutsWrapper GetHostCutMatrix () {
  HistogramCutsWrapper cmat;
  cmat.SetPtrs({0, 3, 6, 9, 12, 15, 18, 21, 24});
  cmat.SetMins({0.1f, 0.2f, 0.3f, 0.1f, 0.2f, 0.3f, 0.2f, 0.2f});
  // 24 cut fields, 3 cut fields for each feature (column).
  // Each row of the cut represents the cuts for a data column.
  cmat.SetValues({0.30f, 0.67f, 1.64f,
              0.32f, 0.77f, 1.95f,
              0.29f, 0.70f, 1.80f,
              0.32f, 0.75f, 1.85f,
              0.18f, 0.59f, 1.69f,
              0.25f, 0.74f, 2.00f,
              0.26f, 0.74f, 1.98f,
              0.26f, 0.71f, 1.83f});
  return cmat;
}

// TODO(trivialfis): This test is over simplified.
TEST(GpuHist, EvaluateSplits) {
  constexpr int kNRows = 16;
  constexpr int kNCols = 8;

  TrainParam param;

  std::vector<std::pair<std::string, std::string>> args {
    {"max_depth", "1"},
    {"max_leaves", "0"},

    // Disable all other parameters.
    {"colsample_bynode", "1"},
    {"colsample_bylevel", "1"},
    {"colsample_bytree", "1"},
    {"min_child_weight", "0.01"},
    {"reg_alpha", "0"},
    {"reg_lambda", "0"},
    {"max_delta_step", "0"}
  };
  param.Init(args);
  for (size_t i = 0; i < kNCols; ++i) {
    param.monotone_constraints.emplace_back(0);
  }

  int max_bins = 4;

  // Initialize GPUHistMakerDevice
  auto page = BuildEllpackPage(kNRows, kNCols);
  GPUHistMakerDevice<GradientPairPrecise> maker(0, page.get(), kNRows, param, kNCols, kNCols);
  // Initialize GPUHistMakerDevice::node_sum_gradients
  maker.node_sum_gradients = {{6.4f, 12.8f}};

  // Initialize GPUHistMakerDevice::cut
  auto cmat = GetHostCutMatrix();

  // Copy cut matrix to device.
  maker.ba.Allocate(0,
                    &(page->matrix.info.feature_segments), cmat.Ptrs().size(),
                    &(page->matrix.info.min_fvalue), cmat.MinValues().size(),
                    &(page->matrix.info.gidx_fvalue_map), 24,
                    &(maker.monotone_constraints), kNCols);
  dh::CopyVectorToDeviceSpan(page->matrix.info.feature_segments, cmat.Ptrs());
  dh::CopyVectorToDeviceSpan(page->matrix.info.gidx_fvalue_map, cmat.Values());
  dh::CopyVectorToDeviceSpan(maker.monotone_constraints, param.monotone_constraints);
  dh::CopyVectorToDeviceSpan(page->matrix.info.min_fvalue, cmat.MinValues());

  // Initialize GPUHistMakerDevice::hist
  maker.hist.Init(0, (max_bins - 1) * kNCols);
  maker.hist.AllocateHistogram(0);
  // Each row of hist_gpair represents gpairs for one feature.
  // Each entry represents a bin.
  std::vector<GradientPairPrecise> hist_gpair = GetHostHistGpair();
  std::vector<bst_float> hist;
  for (auto pair : hist_gpair) {
    hist.push_back(pair.GetGrad());
    hist.push_back(pair.GetHess());
  }

  ASSERT_EQ(maker.hist.Data().size(), hist.size());
  thrust::copy(hist.begin(), hist.end(),
               maker.hist.Data().begin());

  maker.column_sampler.Init(kNCols,
                            param.colsample_bynode,
                            param.colsample_bylevel,
                            param.colsample_bytree,
                            false);

  RegTree tree;
  MetaInfo info;
  info.num_row_ = kNRows;
  info.num_col_ = kNCols;

  maker.node_value_constraints.resize(1);
  maker.node_value_constraints[0].lower_bound = -1.0;
  maker.node_value_constraints[0].upper_bound = 1.0;

  std::vector<DeviceSplitCandidate> res = maker.EvaluateSplits({0, 0 }, tree, kNCols);

  ASSERT_EQ(res[0].findex, 7);
  ASSERT_EQ(res[1].findex, 7);
  ASSERT_NEAR(res[0].fvalue, 0.26, xgboost::kRtEps);
  ASSERT_NEAR(res[1].fvalue, 0.26, xgboost::kRtEps);
}

void TestHistogramIndexImpl() {
  // Test if the compressed histogram index matches when using a sparse
  // dmatrix with and without using external memory

  int constexpr kNRows = 1000, kNCols = 10;

  // Build 2 matrices and build a histogram maker with that
  tree::GPUHistMakerSpecialised<GradientPairPrecise> hist_maker, hist_maker_ext;
  std::unique_ptr<DMatrix> hist_maker_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 0, true));

  dmlc::TemporaryDirectory tempdir;
  std::unique_ptr<DMatrix> hist_maker_ext_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 128UL, true, tempdir));

  std::vector<std::pair<std::string, std::string>> training_params = {
    {"max_depth", "10"},
    {"max_leaves", "0"}
  };

  GenericParameter generic_param(CreateEmptyGenericParam(0));
  hist_maker.Configure(training_params, &generic_param);
  hist_maker.InitDataOnce(hist_maker_dmat.get());
  hist_maker_ext.Configure(training_params, &generic_param);
  hist_maker_ext.InitDataOnce(hist_maker_ext_dmat.get());

  // Extract the device maker from the histogram makers and from that its compressed
  // histogram index
  const auto &maker = hist_maker.maker;
  std::vector<common::CompressedByteT> h_gidx_buffer(maker->page->gidx_buffer.size());
  dh::CopyDeviceSpanToVector(&h_gidx_buffer, maker->page->gidx_buffer);

  const auto &maker_ext = hist_maker_ext.maker;
  std::vector<common::CompressedByteT> h_gidx_buffer_ext(maker_ext->page->gidx_buffer.size());
  dh::CopyDeviceSpanToVector(&h_gidx_buffer_ext, maker_ext->page->gidx_buffer);

  ASSERT_EQ(maker->page->matrix.info.n_bins, maker_ext->page->matrix.info.n_bins);
  ASSERT_EQ(maker->page->gidx_buffer.size(), maker_ext->page->gidx_buffer.size());

  ASSERT_EQ(h_gidx_buffer, h_gidx_buffer_ext);
}

TEST(GpuHist, TestHistogramIndex) {
  TestHistogramIndexImpl();
}

// gamma is an alias of min_split_loss
int32_t TestMinSplitLoss(DMatrix* dmat, float gamma, HostDeviceVector<GradientPair>* gpair) {
  Args args {
    {"max_depth", "1"},
    {"max_leaves", "0"},

    // Disable all other parameters.
    {"colsample_bynode", "1"},
    {"colsample_bylevel", "1"},
    {"colsample_bytree", "1"},
    {"min_child_weight", "0.01"},
    {"reg_alpha", "0"},
    {"reg_lambda", "0"},
    {"max_delta_step", "0"},

    // test gamma
    {"gamma", std::to_string(gamma)}
  };

  tree::GPUHistMakerSpecialised<GradientPairPrecise> hist_maker;
  GenericParameter generic_param(CreateEmptyGenericParam(0));
  hist_maker.Configure(args, &generic_param);

  RegTree tree;
  hist_maker.Update(gpair, dmat, {&tree});

  auto n_nodes = tree.NumExtraNodes();
  return n_nodes;
}

TEST(GpuHist, MinSplitLoss) {
  constexpr size_t kRows = 32;
  constexpr size_t kCols = 16;
  constexpr float kSparsity = 0.6;
  auto dmat = CreateDMatrix(kRows, kCols, kSparsity, 3);

  xgboost::SimpleLCG gen;
  xgboost::SimpleRealUniformDistribution<bst_float> dist(0.0f, 1.0f);
  std::vector<GradientPair> h_gpair(kRows);
  for (auto &gpair : h_gpair) {
    bst_float grad = dist(&gen);
    bst_float hess = dist(&gen);
    gpair = GradientPair(grad, hess);
  }
  HostDeviceVector<GradientPair> gpair(h_gpair);

  {
    int32_t n_nodes = TestMinSplitLoss((*dmat).get(), 0.01, &gpair);
    // This is not strictly verified, meaning the numeber `2` is whatever GPU_Hist retured
    // when writing this test, and only used for testing larger gamma (below) does prevent
    // building tree.
    ASSERT_EQ(n_nodes, 2);
  }
  {
    int32_t n_nodes = TestMinSplitLoss((*dmat).get(), 100.0, &gpair);
    // No new nodes with gamma == 100.
    ASSERT_EQ(n_nodes, static_cast<decltype(n_nodes)>(0));
  }
  delete dmat;
}

}  // namespace tree
}  // namespace xgboost
