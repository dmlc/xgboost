/**
 * Copyright 2017-2023 by XGBoost contributors
 */
#include <gtest/gtest.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <xgboost/base.h>

#include <random>
#include <string>
#include <vector>

#include "../../../src/common/common.h"
#include "../../../src/data/ellpack_page.cuh"  // for EllpackPageImpl
#include "../../../src/data/ellpack_page.h"    // for EllpackPage
#include "../../../src/tree/param.h"  // for TrainParam
#include "../../../src/tree/updater_gpu_hist.cu"
#include "../filesystem.h"  // dmlc::TemporaryDirectory
#include "../helpers.h"
#include "../histogram_helpers.h"
#include "xgboost/context.h"
#include "xgboost/json.h"

namespace xgboost::tree {
TEST(GpuHist, DeviceHistogram) {
  // Ensures that node allocates correctly after reaching `kStopGrowingSize`.
  dh::safe_cuda(hipSetDevice(0));
  constexpr size_t kNBins = 128;
  constexpr int kNNodes = 4;
  constexpr size_t kStopGrowing = kNNodes * kNBins * 2u;
  DeviceHistogramStorage<kStopGrowing> histogram;
  histogram.Init(FstCU(), kNBins);
  for (int i = 0; i < kNNodes; ++i) {
    histogram.AllocateHistograms({i});
  }
  histogram.Reset();
  ASSERT_EQ(histogram.Data().size(), kStopGrowing);

  // Use allocated memory but do not erase nidx_map.
  for (int i = 0; i < kNNodes; ++i) {
    histogram.AllocateHistograms({i});
  }
  for (int i = 0; i < kNNodes; ++i) {
    ASSERT_TRUE(histogram.HistogramExists(i));
  }

  // Add two new nodes
  histogram.AllocateHistograms({kNNodes});
  histogram.AllocateHistograms({kNNodes + 1});

  // Old cached nodes should still exist
  for (int i = 0; i < kNNodes; ++i) {
    ASSERT_TRUE(histogram.HistogramExists(i));
  }

  // Should be deleted
  ASSERT_FALSE(histogram.HistogramExists(kNNodes));
  // Most recent node should exist
  ASSERT_TRUE(histogram.HistogramExists(kNNodes + 1));

  // Add same node again - should fail
  EXPECT_ANY_THROW(histogram.AllocateHistograms({kNNodes + 1}););
}

std::vector<GradientPairPrecise> GetHostHistGpair() {
  // 24 bins, 3 bins for each feature (column).
  std::vector<GradientPairPrecise> hist_gpair = {
    {0.8314f, 0.7147f}, {1.7989f, 3.7312f}, {3.3846f, 3.4598f},
    {2.9277f, 3.5886f}, {1.8429f, 2.4152f}, {1.2443f, 1.9019f},
    {1.6380f, 2.9174f}, {1.5657f, 2.5107f}, {2.8111f, 2.4776f},
    {2.1322f, 3.0651f}, {3.2927f, 3.8540f}, {0.5899f, 0.9866f},
    {1.5185f, 1.6263f}, {2.0686f, 3.1844f}, {2.4278f, 3.0950f},
    {1.5105f, 2.1403f}, {2.6922f, 4.2217f}, {1.8122f, 1.5437f},
    {0.0000f, 0.0000f}, {4.3245f, 5.7955f}, {1.6903f, 2.1103f},
    {2.4012f, 4.4754f}, {3.6136f, 3.4303f}, {0.0000f, 0.0000f}
  };
  return hist_gpair;
}

template <typename GradientSumT>
void TestBuildHist(bool use_shared_memory_histograms) {
  int const kNRows = 16, kNCols = 8;

  TrainParam param;
  Args args{
      {"max_depth", "6"},
      {"max_leaves", "0"},
  };
  param.Init(args);

  auto page = BuildEllpackPage(kNRows, kNCols);
  BatchParam batch_param{};
  Context ctx{MakeCUDACtx(0)};
  auto cs = std::make_shared<common::ColumnSampler>(0);
  GPUHistMakerDevice maker(&ctx, /*is_external_memory=*/false, {}, kNRows, param, cs, kNCols,
                           batch_param, MetaInfo());
  xgboost::SimpleLCG gen;
  xgboost::SimpleRealUniformDistribution<bst_float> dist(0.0f, 1.0f);
  HostDeviceVector<GradientPair> gpair(kNRows);
  for (auto &gp : gpair.HostVector()) {
    bst_float grad = dist(&gen);
    bst_float hess = dist(&gen);
    gp = GradientPair(grad, hess);
  }
  gpair.SetDevice(DeviceOrd::CUDA(0));

  thrust::host_vector<common::CompressedByteT> h_gidx_buffer (page->gidx_buffer.HostVector());
  maker.row_partitioner = std::make_unique<RowPartitioner>(FstCU(), kNRows);

  maker.hist.Init(FstCU(), page->Cuts().TotalBins());
  maker.hist.AllocateHistograms({0});

  maker.gpair = gpair.DeviceSpan();
  maker.quantiser = std::make_unique<GradientQuantiser>(&ctx, maker.gpair, MetaInfo());
  maker.page = page.get();

  maker.InitFeatureGroupsOnce();

  BuildGradientHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(DeviceOrd::CUDA(0)),
                         maker.feature_groups->DeviceAccessor(DeviceOrd::CUDA(0)), gpair.DeviceSpan(),
                         maker.row_partitioner->GetRows(0), maker.hist.GetNodeHistogram(0),
                         *maker.quantiser, !use_shared_memory_histograms);

  DeviceHistogramStorage<>& d_hist = maker.hist;

  auto node_histogram = d_hist.GetNodeHistogram(0);
  // d_hist.data stored in float, not gradient pair
  thrust::host_vector<GradientPairInt64> h_result (node_histogram.size());
  dh::safe_cuda(hipMemcpy(h_result.data(), node_histogram.data(), node_histogram.size_bytes(),
                           hipMemcpyDeviceToHost));

  std::vector<GradientPairPrecise> solution = GetHostHistGpair();
  for (size_t i = 0; i < h_result.size(); ++i) {
    auto result = maker.quantiser->ToFloatingPoint(h_result[i]);
    ASSERT_NEAR(result.GetGrad(), solution[i].GetGrad(), 0.01f);
    ASSERT_NEAR(result.GetHess(), solution[i].GetHess(), 0.01f);
  }
}

TEST(GpuHist, BuildHistGlobalMem) {
  TestBuildHist<GradientPairPrecise>(false);
}

TEST(GpuHist, BuildHistSharedMem) {
  TestBuildHist<GradientPairPrecise>(true);
}

HistogramCutsWrapper GetHostCutMatrix () {
  HistogramCutsWrapper cmat;
  cmat.SetPtrs({0, 3, 6, 9, 12, 15, 18, 21, 24});
  cmat.SetMins({0.1f, 0.2f, 0.3f, 0.1f, 0.2f, 0.3f, 0.2f, 0.2f});
  // 24 cut fields, 3 cut fields for each feature (column).
  // Each row of the cut represents the cuts for a data column.
  cmat.SetValues({0.30f, 0.67f, 1.64f,
              0.32f, 0.77f, 1.95f,
              0.29f, 0.70f, 1.80f,
              0.32f, 0.75f, 1.85f,
              0.18f, 0.59f, 1.69f,
              0.25f, 0.74f, 2.00f,
              0.26f, 0.74f, 1.98f,
              0.26f, 0.71f, 1.83f});
  return cmat;
}

void TestHistogramIndexImpl() {
  // Test if the compressed histogram index matches when using a sparse
  // dmatrix with and without using external memory

  int constexpr kNRows = 1000, kNCols = 10;

  // Build 2 matrices and build a histogram maker with that
  Context ctx(MakeCUDACtx(0));
  ObjInfo task{ObjInfo::kRegression};
  tree::GPUHistMaker hist_maker{&ctx, &task}, hist_maker_ext{&ctx, &task};
  std::unique_ptr<DMatrix> hist_maker_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 0, true));

  dmlc::TemporaryDirectory tempdir;
  std::unique_ptr<DMatrix> hist_maker_ext_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 128UL, true, tempdir));

  Args training_params = {{"max_depth", "10"}, {"max_leaves", "0"}};
  TrainParam param;
  param.UpdateAllowUnknown(training_params);

  hist_maker.Configure(training_params);
  hist_maker.InitDataOnce(&param, hist_maker_dmat.get());
  hist_maker_ext.Configure(training_params);
  hist_maker_ext.InitDataOnce(&param, hist_maker_ext_dmat.get());

  // Extract the device maker from the histogram makers and from that its compressed
  // histogram index
  const auto &maker = hist_maker.maker;
  auto grad = GenerateRandomGradients(kNRows);
  grad.SetDevice(DeviceOrd::CUDA(0));
  maker->Reset(&grad, hist_maker_dmat.get(), kNCols);
  std::vector<common::CompressedByteT> h_gidx_buffer(maker->page->gidx_buffer.HostVector());

  const auto &maker_ext = hist_maker_ext.maker;
  maker_ext->Reset(&grad, hist_maker_ext_dmat.get(), kNCols);
  std::vector<common::CompressedByteT> h_gidx_buffer_ext(maker_ext->page->gidx_buffer.HostVector());

  ASSERT_EQ(maker->page->Cuts().TotalBins(), maker_ext->page->Cuts().TotalBins());
  ASSERT_EQ(maker->page->gidx_buffer.Size(), maker_ext->page->gidx_buffer.Size());
}

TEST(GpuHist, TestHistogramIndex) {
  TestHistogramIndexImpl();
}

void UpdateTree(Context const* ctx, linalg::Matrix<GradientPair>* gpair, DMatrix* dmat,
                size_t gpu_page_size, RegTree* tree, HostDeviceVector<bst_float>* preds,
                float subsample = 1.0f, const std::string& sampling_method = "uniform",
                int max_bin = 2) {
  if (gpu_page_size > 0) {
    // Loop over the batches and count the records
    int64_t batch_count = 0;
    int64_t row_count = 0;
    for (const auto& batch : dmat->GetBatches<EllpackPage>(
             ctx, BatchParam{max_bin, TrainParam::DftSparseThreshold()})) {
      EXPECT_LT(batch.Size(), dmat->Info().num_row_);
      batch_count++;
      row_count += batch.Size();
    }
    EXPECT_GE(batch_count, 2);
    EXPECT_EQ(row_count, dmat->Info().num_row_);
  }

  Args args{
      {"max_depth", "2"},
      {"max_bin", std::to_string(max_bin)},
      {"min_child_weight", "0.0"},
      {"reg_alpha", "0"},
      {"reg_lambda", "0"},
      {"subsample", std::to_string(subsample)},
      {"sampling_method", sampling_method},
  };
  TrainParam param;
  param.UpdateAllowUnknown(args);

  ObjInfo task{ObjInfo::kRegression};
  tree::GPUHistMaker hist_maker{ctx, &task};
  hist_maker.Configure(Args{});

  std::vector<HostDeviceVector<bst_node_t>> position(1);
  hist_maker.Update(&param, gpair, dmat, common::Span<HostDeviceVector<bst_node_t>>{position},
                    {tree});
  auto cache = linalg::MakeTensorView(ctx, preds->DeviceSpan(), preds->Size(), 1);
  hist_maker.UpdatePredictionCache(dmat, cache);
}

TEST(GpuHist, UniformSampling) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr float kSubsample = 0.9999;
  common::GlobalRandom().seed(1994);

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  linalg::Matrix<GradientPair> gpair({kRows}, Context{}.MakeCUDA().Device());
  gpair.Data()->Copy(GenerateRandomGradients(kRows));

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, DeviceOrd::CUDA(0));
  Context ctx(MakeCUDACtx(0));
  UpdateTree(&ctx, &gpair, dmat.get(), 0, &tree, &preds, 1.0, "uniform", kRows);
  // Build another tree using sampling.
  RegTree tree_sampling;
  HostDeviceVector<bst_float> preds_sampling(kRows, 0.0, DeviceOrd::CUDA(0));
  UpdateTree(&ctx, &gpair, dmat.get(), 0, &tree_sampling, &preds_sampling, kSubsample, "uniform",
             kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_sampling_h = preds_sampling.ConstHostVector();
  for (size_t i = 0; i < kRows; i++) {
    EXPECT_NEAR(preds_h[i], preds_sampling_h[i], 1e-8);
  }
}

TEST(GpuHist, GradientBasedSampling) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr float kSubsample = 0.9999;
  common::GlobalRandom().seed(1994);

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  linalg::Matrix<GradientPair> gpair({kRows}, MakeCUDACtx(0).Device());
  gpair.Data()->Copy(GenerateRandomGradients(kRows));

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, DeviceOrd::CUDA(0));
  Context ctx(MakeCUDACtx(0));
  UpdateTree(&ctx, &gpair, dmat.get(), 0, &tree, &preds, 1.0, "uniform", kRows);

  // Build another tree using sampling.
  RegTree tree_sampling;
  HostDeviceVector<bst_float> preds_sampling(kRows, 0.0, DeviceOrd::CUDA(0));
  UpdateTree(&ctx, &gpair, dmat.get(), 0, &tree_sampling, &preds_sampling, kSubsample,
             "gradient_based", kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_sampling_h = preds_sampling.ConstHostVector();
  for (size_t i = 0; i < kRows; i++) {
    EXPECT_NEAR(preds_h[i], preds_sampling_h[i], 1e-3);
  }
}

TEST(GpuHist, ExternalMemory) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr size_t kPageSize = 1024;

  dmlc::TemporaryDirectory tmpdir;

  // Create a DMatrix with multiple batches.
  std::unique_ptr<DMatrix> dmat_ext(
      CreateSparsePageDMatrix(kRows, kCols, kRows / kPageSize, tmpdir.path + "/cache"));

  // Create a single batch DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrix(kRows, kCols, 1, tmpdir.path + "/cache"));

  Context ctx(MakeCUDACtx(0));
  linalg::Matrix<GradientPair> gpair({kRows}, ctx.Device());
  gpair.Data()->Copy(GenerateRandomGradients(kRows));

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, DeviceOrd::CUDA(0));
  UpdateTree(&ctx, &gpair, dmat.get(), 0, &tree, &preds, 1.0, "uniform", kRows);
  // Build another tree using multiple ELLPACK pages.
  RegTree tree_ext;
  HostDeviceVector<bst_float> preds_ext(kRows, 0.0, DeviceOrd::CUDA(0));
  UpdateTree(&ctx, &gpair, dmat_ext.get(), kPageSize, &tree_ext, &preds_ext, 1.0, "uniform", kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_ext_h = preds_ext.ConstHostVector();
  for (size_t i = 0; i < kRows; i++) {
    EXPECT_NEAR(preds_h[i], preds_ext_h[i], 1e-6);
  }
}

TEST(GpuHist, ExternalMemoryWithSampling) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr size_t kPageSize = 1024;
  constexpr float kSubsample = 0.5;
  const std::string kSamplingMethod = "gradient_based";
  common::GlobalRandom().seed(0);

  dmlc::TemporaryDirectory tmpdir;

  // Create a single batch DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrix(kRows, kCols, 1, tmpdir.path + "/cache"));

  // Create a DMatrix with multiple batches.
  std::unique_ptr<DMatrix> dmat_ext(
      CreateSparsePageDMatrix(kRows, kCols, kRows / kPageSize, tmpdir.path + "/cache"));

  Context ctx(MakeCUDACtx(0));
  linalg::Matrix<GradientPair> gpair({kRows}, ctx.Device());
  gpair.Data()->Copy(GenerateRandomGradients(kRows));

  // Build a tree using the in-memory DMatrix.
  auto rng = common::GlobalRandom();

  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, DeviceOrd::CUDA(0));
  UpdateTree(&ctx, &gpair, dmat.get(), 0, &tree, &preds, kSubsample, kSamplingMethod, kRows);

  // Build another tree using multiple ELLPACK pages.
  common::GlobalRandom() = rng;
  RegTree tree_ext;
  HostDeviceVector<bst_float> preds_ext(kRows, 0.0, DeviceOrd::CUDA(0));
  UpdateTree(&ctx, &gpair, dmat_ext.get(), kPageSize, &tree_ext, &preds_ext, kSubsample,
             kSamplingMethod, kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_ext_h = preds_ext.ConstHostVector();
  for (size_t i = 0; i < kRows; i++) {
    ASSERT_NEAR(preds_h[i], preds_ext_h[i], 1e-3);
  }
}

TEST(GpuHist, ConfigIO) {
  Context ctx(MakeCUDACtx(0));
  ObjInfo task{ObjInfo::kRegression};
  std::unique_ptr<TreeUpdater> updater{TreeUpdater::Create("grow_gpu_hist", &ctx, &task)};
  updater->Configure(Args{});

  Json j_updater{Object{}};
  updater->SaveConfig(&j_updater);
  ASSERT_TRUE(IsA<Object>(j_updater["hist_train_param"]));
  updater->LoadConfig(j_updater);

  Json j_updater_roundtrip{Object{}};
  updater->SaveConfig(&j_updater_roundtrip);
  ASSERT_TRUE(IsA<Object>(j_updater_roundtrip["hist_train_param"]));

  ASSERT_EQ(j_updater, j_updater_roundtrip);
}

TEST(GpuHist, MaxDepth) {
  Context ctx(MakeCUDACtx(0));
  size_t constexpr kRows = 16;
  size_t constexpr kCols = 4;
  auto p_mat = RandomDataGenerator{kRows, kCols, 0}.GenerateDMatrix();

  auto learner = std::unique_ptr<Learner>(Learner::Create({p_mat}));
  learner->SetParam("max_depth", "32");
  learner->Configure();

  ASSERT_THROW({learner->UpdateOneIter(0, p_mat);}, dmlc::Error);
}

namespace {
RegTree GetHistTree(Context const* ctx, DMatrix* dmat) {
  ObjInfo task{ObjInfo::kRegression};
  GPUHistMaker hist_maker{ctx, &task};
  hist_maker.Configure(Args{});

  TrainParam param;
  param.UpdateAllowUnknown(Args{});

  linalg::Matrix<GradientPair> gpair({dmat->Info().num_row_}, ctx->Device());
  gpair.Data()->Copy(GenerateRandomGradients(dmat->Info().num_row_));

  std::vector<HostDeviceVector<bst_node_t>> position(1);
  RegTree tree;
  hist_maker.Update(&param, &gpair, dmat, common::Span<HostDeviceVector<bst_node_t>>{position},
                    {&tree});
  return tree;
}

void VerifyHistColumnSplit(bst_row_t rows, bst_feature_t cols, RegTree const& expected_tree) {
  Context ctx(MakeCUDACtx(GPUIDX));

  auto Xy = RandomDataGenerator{rows, cols, 0}.GenerateDMatrix(true);
  auto const world_size = collective::GetWorldSize();
  auto const rank = collective::GetRank();
  std::unique_ptr<DMatrix> sliced{Xy->SliceCol(world_size, rank)};

  RegTree tree = GetHistTree(&ctx, sliced.get());

  Json json{Object{}};
  tree.SaveModel(&json);
  Json expected_json{Object{}};
  expected_tree.SaveModel(&expected_json);
  ASSERT_EQ(json, expected_json);
}
}  // anonymous namespace

class MGPUHistTest : public BaseMGPUTest {};

TEST_F(MGPUHistTest, GPUHistColumnSplit) {
  auto constexpr kRows = 32;
  auto constexpr kCols = 16;

  Context ctx(MakeCUDACtx(0));
  auto dmat = RandomDataGenerator{kRows, kCols, 0}.GenerateDMatrix(true);
  RegTree expected_tree = GetHistTree(&ctx, dmat.get());

  DoTest(VerifyHistColumnSplit, kRows, kCols, expected_tree);
}

namespace {
RegTree GetApproxTree(Context const* ctx, DMatrix* dmat) {
  ObjInfo task{ObjInfo::kRegression};
  GPUGlobalApproxMaker approx_maker{ctx, &task};
  approx_maker.Configure(Args{});

  TrainParam param;
  param.UpdateAllowUnknown(Args{});

  linalg::Matrix<GradientPair> gpair({dmat->Info().num_row_}, ctx->Device());
  gpair.Data()->Copy(GenerateRandomGradients(dmat->Info().num_row_));

  std::vector<HostDeviceVector<bst_node_t>> position(1);
  RegTree tree;
  approx_maker.Update(&param, &gpair, dmat, common::Span<HostDeviceVector<bst_node_t>>{position},
                      {&tree});
  return tree;
}

void VerifyApproxColumnSplit(bst_row_t rows, bst_feature_t cols, RegTree const& expected_tree) {
  Context ctx(MakeCUDACtx(GPUIDX));

  auto Xy = RandomDataGenerator{rows, cols, 0}.GenerateDMatrix(true);
  auto const world_size = collective::GetWorldSize();
  auto const rank = collective::GetRank();
  std::unique_ptr<DMatrix> sliced{Xy->SliceCol(world_size, rank)};

  RegTree tree = GetApproxTree(&ctx, sliced.get());

  Json json{Object{}};
  tree.SaveModel(&json);
  Json expected_json{Object{}};
  expected_tree.SaveModel(&expected_json);
  ASSERT_EQ(json, expected_json);
}
}  // anonymous namespace

class MGPUApproxTest : public BaseMGPUTest {};

TEST_F(MGPUApproxTest, GPUApproxColumnSplit) {
  auto constexpr kRows = 32;
  auto constexpr kCols = 16;

  Context ctx(MakeCUDACtx(0));
  auto dmat = RandomDataGenerator{kRows, kCols, 0}.GenerateDMatrix(true);
  RegTree expected_tree = GetApproxTree(&ctx, dmat.get());

  DoTest(VerifyApproxColumnSplit, kRows, kCols, expected_tree);
}
}  // namespace xgboost::tree
