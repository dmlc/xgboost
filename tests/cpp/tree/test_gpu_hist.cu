/*!
 * Copyright 2017-2018 XGBoost contributors
 */

#include <thrust/device_vector.h>
#include <xgboost/base.h>
#include <random>
#include "../helpers.h"
#include "gtest/gtest.h"

#include "../../../src/data/sparse_page_source.h"
#include "../../../src/gbm/gbtree_model.h"
#include "../../../src/tree/updater_gpu_hist.cu"
#include "../../../src/tree/updater_gpu_common.cuh"
#include "../../../src/common/common.h"

namespace xgboost {
namespace tree {

void BuildGidx(DeviceShard* shard, int n_rows, int n_cols,
               bst_float sparsity=0) {
  auto dmat = CreateDMatrix(n_rows, n_cols, sparsity, 3);
  const SparsePage& batch = *(*dmat)->GetRowBatches().begin();

  common::HistCutMatrix cmat;
  cmat.row_ptr = {0, 3, 6, 9, 12, 15, 18, 21, 24};
  cmat.min_val = {0.1, 0.2, 0.3, 0.1, 0.2, 0.3, 0.2, 0.2};
  // 24 cut fields, 3 cut fields for each feature (column).
  cmat.cut = {0.30, 0.67, 1.64,
              0.32, 0.77, 1.95,
              0.29, 0.70, 1.80,
              0.32, 0.75, 1.85,
              0.18, 0.59, 1.69,
              0.25, 0.74, 2.00,
              0.26, 0.74, 1.98,
              0.26, 0.71, 1.83};

  shard->InitRowPtrs(batch);
  shard->InitCompressedData(cmat, batch);

  delete dmat;
}

TEST(GpuHist, BuildGidxDense) {
  int const n_rows = 16, n_cols = 8;
  TrainParam param;
  param.max_depth = 1;
  param.n_gpus = 1;
  param.max_leaves = 0;

  DeviceShard shard(0, 0, 0, n_rows, param);
  BuildGidx(&shard, n_rows, n_cols);

  std::vector<common::CompressedByteT> h_gidx_buffer;
  h_gidx_buffer = shard.gidx_buffer.AsVector();
  common::CompressedIterator<uint32_t> gidx(h_gidx_buffer.data(), 25);

  ASSERT_EQ(shard.row_stride, n_cols);

  std::vector<uint32_t> solution = {
    0, 3, 8,  9, 14, 17, 20, 21,
    0, 4, 7, 10, 14, 16, 19, 22,
    1, 3, 7, 11, 14, 15, 19, 21,
    2, 3, 7,  9, 13, 16, 20, 22,
    2, 3, 6,  9, 12, 16, 20, 21,
    1, 5, 6, 10, 13, 16, 20, 21,
    2, 5, 8,  9, 13, 17, 19, 22,
    2, 4, 6, 10, 14, 17, 19, 21,
    2, 5, 7,  9, 13, 16, 19, 22,
    0, 3, 8, 10, 12, 16, 19, 22,
    1, 3, 7, 10, 13, 16, 19, 21,
    1, 3, 8, 10, 13, 17, 20, 22,
    2, 4, 6,  9, 14, 15, 19, 22,
    1, 4, 6,  9, 13, 16, 19, 21,
    2, 4, 8, 10, 14, 15, 19, 22,
    1, 4, 7, 10, 14, 16, 19, 21,
  };
  for (size_t i = 0; i < n_rows * n_cols; ++i) {
    ASSERT_EQ(solution[i], gidx[i]);
  }
}

TEST(GpuHist, BuildGidxSparse) {
  int const n_rows = 16, n_cols = 8;
  TrainParam param;
  param.max_depth = 1;
  param.n_gpus = 1;
  param.max_leaves = 0;

  DeviceShard shard(0, 0, 0, n_rows, param);
  BuildGidx(&shard, n_rows, n_cols, 0.9f);

  std::vector<common::CompressedByteT> h_gidx_buffer;
  h_gidx_buffer = shard.gidx_buffer.AsVector();
  common::CompressedIterator<uint32_t> gidx(h_gidx_buffer.data(), 25);

  ASSERT_LE(shard.row_stride, 3);

  // row_stride = 3, 16 rows, 48 entries for ELLPack
  std::vector<uint32_t> solution = {
    15, 24, 24,  0, 24, 24, 24, 24, 24, 24, 24, 24, 20, 24, 24, 24,
    24, 24, 24, 24, 24,  5, 24, 24,  0, 16, 24, 15, 24, 24, 24, 24,
    24,  7, 14, 16,  4, 24, 24, 24, 24, 24,  9, 24, 24,  1, 24, 24
  };
  for (size_t i = 0; i < n_rows * shard.row_stride; ++i) {
    ASSERT_EQ(solution[i], gidx[i]);
  }
}

std::vector<GradientPairPrecise> GetHostHistGpair() {
  // 24 bins, 3 bins for each feature (column).
  std::vector<GradientPairPrecise> hist_gpair = {
    {0.8314, 0.7147}, {1.7989, 3.7312}, {3.3846, 3.4598},
    {2.9277, 3.5886}, {1.8429, 2.4152}, {1.2443, 1.9019},
    {1.6380, 2.9174}, {1.5657, 2.5107}, {2.8111, 2.4776},
    {2.1322, 3.0651}, {3.2927, 3.8540}, {0.5899, 0.9866},
    {1.5185, 1.6263}, {2.0686, 3.1844}, {2.4278, 3.0950},
    {1.5105, 2.1403}, {2.6922, 4.2217}, {1.8122, 1.5437},
    {0.0000, 0.0000}, {4.3245, 5.7955}, {1.6903, 2.1103},
    {2.4012, 4.4754}, {3.6136, 3.4303}, {0.0000, 0.0000}
  };
  return hist_gpair;
}

void TestBuildHist(GPUHistBuilderBase& builder) {
  int const n_rows = 16, n_cols = 8;

  TrainParam param;
  param.max_depth = 6;
  param.n_gpus = 1;
  param.max_leaves = 0;

  DeviceShard shard(0, 0, 0, n_rows, param);

  BuildGidx(&shard, n_rows, n_cols);

  xgboost::SimpleLCG gen;
  xgboost::SimpleRealUniformDistribution<bst_float> dist(0.0f, 1.0f);
  std::vector<GradientPair> h_gpair(n_rows);
  for (size_t i = 0; i < h_gpair.size(); ++i) {
    bst_float grad = dist(&gen);
    bst_float hess = dist(&gen);
    h_gpair[i] = GradientPair(grad, hess);
  }

  thrust::device_vector<GradientPair> gpair (n_rows);
  gpair = h_gpair;

  int num_symbols = shard.n_bins + 1;

  thrust::host_vector<common::CompressedByteT> h_gidx_buffer (
      shard.gidx_buffer.Size());

  common::CompressedByteT* d_gidx_buffer_ptr = shard.gidx_buffer.Data();
  dh::safe_cuda(hipMemcpy(h_gidx_buffer.data(), d_gidx_buffer_ptr,
                           sizeof(common::CompressedByteT) * shard.gidx_buffer.Size(),
                           hipMemcpyDeviceToHost));
  auto gidx = common::CompressedIterator<uint32_t>(h_gidx_buffer.data(),
                                                   num_symbols);

  shard.ridx_segments.resize(1);
  shard.ridx_segments[0] = Segment(0, n_rows);
  shard.hist.AllocateHistogram(0);
  shard.gpair.copy(gpair.begin(), gpair.end());
  thrust::sequence(shard.ridx.CurrentDVec().tbegin(),
                   shard.ridx.CurrentDVec().tend());

  builder.Build(&shard, 0);
  DeviceHistogram d_hist = shard.hist;

  GradientPairSumT* d_histptr {d_hist.GetHistPtr(0)};
  // d_hist.data stored in float, not gradient pair
  thrust::host_vector<GradientPairSumT> h_result (d_hist.data.size()/2);
  size_t data_size = sizeof(GradientPairSumT) / (
      sizeof(GradientPairSumT) / sizeof(GradientPairSumT::ValueT));
  data_size *= d_hist.data.size();
  dh::safe_cuda(hipMemcpy(h_result.data(), d_histptr, data_size,
                           hipMemcpyDeviceToHost));

  std::vector<GradientPairPrecise> solution = GetHostHistGpair();
  std::cout << std::fixed;
  for (size_t i = 0; i < h_result.size(); ++i) {
    EXPECT_NEAR(h_result[i].GetGrad(), solution[i].GetGrad(), 0.01f);
    EXPECT_NEAR(h_result[i].GetHess(), solution[i].GetHess(), 0.01f);
  }
}

TEST(GpuHist, BuildHistGlobalMem) {
  GlobalMemHistBuilder builder;
  TestBuildHist(builder);
}

TEST(GpuHist, BuildHistSharedMem) {
  SharedMemHistBuilder builder;
  TestBuildHist(builder);
}

common::HistCutMatrix GetHostCutMatrix () {
  common::HistCutMatrix cmat;
  cmat.row_ptr = {0, 3, 6, 9, 12, 15, 18, 21, 24};
  cmat.min_val = {0.1, 0.2, 0.3, 0.1, 0.2, 0.3, 0.2, 0.2};
  // 24 cut fields, 3 cut fields for each feature (column).
  // Each row of the cut represents the cuts for a data column.
  cmat.cut = {0.30, 0.67, 1.64,
              0.32, 0.77, 1.95,
              0.29, 0.70, 1.80,
              0.32, 0.75, 1.85,
              0.18, 0.59, 1.69,
              0.25, 0.74, 2.00,
              0.26, 0.74, 1.98,
              0.26, 0.71, 1.83};
  return cmat;
}

// TODO(trivialfis): This test is over simplified.
TEST(GpuHist, EvaluateSplits) {
  constexpr int n_rows = 16;
  constexpr int n_cols = 8;

  TrainParam param;
  param.max_depth = 1;
  param.n_gpus = 1;
  param.colsample_bylevel = 1;
  param.colsample_bytree = 1;
  param.min_child_weight = 0.01;

  // Disable all parameters.
  param.reg_alpha = 0.0;
  param.reg_lambda = 0;
  param.max_delta_step = 0.0;

  for (size_t i = 0; i < n_cols; ++i) {
    param.monotone_constraints.emplace_back(0);
  }

  int max_bins = 4;

  // Initialize DeviceShard
  std::unique_ptr<DeviceShard> shard {new DeviceShard(0, 0, 0, n_rows, param)};
  // Initialize DeviceShard::node_sum_gradients
  shard->node_sum_gradients = {{6.4, 12.8}};

  // Initialize DeviceShard::cut
  common::HistCutMatrix cmat = GetHostCutMatrix();

  // Copy cut matrix to device.
  DeviceShard::DeviceHistCutMatrix cut;
  shard->ba.Allocate(0, true,
                     &(shard->cut_.feature_segments), cmat.row_ptr.size(),
                     &(shard->cut_.min_fvalue), cmat.min_val.size(),
                     &(shard->cut_.gidx_fvalue_map), 24,
                     &(shard->monotone_constraints), n_cols);
  shard->cut_.feature_segments.copy(cmat.row_ptr.begin(), cmat.row_ptr.end());
  shard->cut_.gidx_fvalue_map.copy(cmat.cut.begin(), cmat.cut.end());
  shard->monotone_constraints.copy(param.monotone_constraints.begin(),
                                   param.monotone_constraints.end());

  // Initialize DeviceShard::hist
  shard->hist.Init(0, (max_bins - 1) * n_cols);
  shard->hist.AllocateHistogram(0);
  // Each row of hist_gpair represents gpairs for one feature.
  // Each entry represents a bin.
  std::vector<GradientPairPrecise> hist_gpair = GetHostHistGpair();
  std::vector<bst_float> hist;
  for (auto pair : hist_gpair) {
    hist.push_back(pair.GetGrad());
    hist.push_back(pair.GetHess());
  }

  ASSERT_EQ(shard->hist.data.size(), hist.size());
  thrust::copy(hist.begin(), hist.end(),
               shard->hist.data.begin());


  // Initialize GPUHistMaker
  GPUHistMaker hist_maker = GPUHistMaker();
  hist_maker.param_ = param;
  hist_maker.shards_.push_back(std::move(shard));
  hist_maker.column_sampler_.Init(n_cols,
                                  param.colsample_bylevel,
                                  param.colsample_bytree,
                                  false);

  RegTree tree;
  tree.InitModel();

  MetaInfo info;
  info.num_row_ = n_rows;
  info.num_col_ = n_cols;

  hist_maker.info_ = &info;
  hist_maker.node_value_constraints_.resize(1);
  hist_maker.node_value_constraints_[0].lower_bound = -1.0;
  hist_maker.node_value_constraints_[0].upper_bound = 1.0;

  std::vector<DeviceSplitCandidate> res =
      hist_maker.EvaluateSplits({0}, &tree);

  ASSERT_EQ(res.size(), 1);
  ASSERT_EQ(res[0].findex, 7);
  ASSERT_NEAR(res[0].fvalue, 0.26, xgboost::kRtEps);
}

TEST(GpuHist, ApplySplit) {
  GPUHistMaker hist_maker = GPUHistMaker();
  int constexpr nid = 0;
  int constexpr n_rows = 16;
  int constexpr n_cols = 8;

  TrainParam param;
  param.silent = true;

  // Initialize shard
  for (size_t i = 0; i < n_cols; ++i) {
    param.monotone_constraints.emplace_back(0);
  }

  hist_maker.shards_.resize(1);
  hist_maker.shards_[0].reset(new DeviceShard(0, 0, 0, n_rows, param));

  auto& shard = hist_maker.shards_.at(0);
  shard->ridx_segments.resize(3);  // 3 nodes.
  shard->node_sum_gradients.resize(3);

  shard->ridx_segments[0] = Segment(0, n_rows);
  shard->ba.Allocate(0, true, &(shard->ridx), n_rows,
                     &(shard->position), n_rows);
  shard->row_stride = n_cols;
  thrust::sequence(shard->ridx.CurrentDVec().tbegin(),
                   shard->ridx.CurrentDVec().tend());
  dh::safe_cuda(hipHostMalloc(&(shard->tmp_pinned), sizeof(int64_t)));

  // Initialize GPUHistMaker
  hist_maker.param_ = param;
  RegTree tree;
  tree.InitModel();

  DeviceSplitCandidate candidate;
  candidate.Update(2, kLeftDir,
                   0.59, 4,  // fvalue has to be equal to one of the cut field
                   GradientPair(8.2, 2.8), GradientPair(6.3, 3.6),
                   GPUTrainingParam(param));
  GPUHistMaker::ExpandEntry candidate_entry {0, 0, candidate, 0};
  candidate_entry.nid = nid;

  auto const& nodes = tree.GetNodes();
  size_t n_nodes = nodes.size();

  // Used to get bin_id in update position.
  common::HistCutMatrix cmat = GetHostCutMatrix();
  hist_maker.hmat_ = cmat;

  MetaInfo info;
  info.num_row_ = n_rows;
  info.num_col_ = n_cols;
  info.num_nonzero_ = n_rows * n_cols;  // Dense

  // Initialize gidx
  int n_bins = 24;
  int row_stride = n_cols;
  int num_symbols = n_bins + 1;
  size_t compressed_size_bytes =
      common::CompressedBufferWriter::CalculateBufferSize(
          row_stride * n_rows, num_symbols);
  shard->ba.Allocate(0, param.silent,
                     &(shard->gidx_buffer), compressed_size_bytes);

  common::CompressedBufferWriter wr(num_symbols);
  std::vector<int> h_gidx (n_rows * row_stride);
  std::iota(h_gidx.begin(), h_gidx.end(), 0);
  std::vector<common::CompressedByteT> h_gidx_compressed (compressed_size_bytes);

  wr.Write(h_gidx_compressed.data(), h_gidx.begin(), h_gidx.end());
  shard->gidx_buffer.copy(h_gidx_compressed.begin(), h_gidx_compressed.end());

  shard->gidx = common::CompressedIterator<uint32_t>(
      shard->gidx_buffer.Data(), num_symbols);

  hist_maker.info_ = &info;
  hist_maker.ApplySplit(candidate_entry, &tree);

  ASSERT_FALSE(tree[nid].IsLeaf());

  int left_nidx = tree[nid].LeftChild();
  int right_nidx = tree[nid].RightChild();

  ASSERT_EQ(shard->ridx_segments[left_nidx].begin, 0);
  ASSERT_EQ(shard->ridx_segments[left_nidx].end, 6);
  ASSERT_EQ(shard->ridx_segments[right_nidx].begin, 6);
  ASSERT_EQ(shard->ridx_segments[right_nidx].end, 16);
}

TEST(GpuHist, MGPU_mock) {
  // Attempt to choose multiple GPU devices
  int ngpu;
  dh::safe_cuda(hipGetDeviceCount(&ngpu));
  CHECK_GT(ngpu, 1);
  for (int i = 0; i < ngpu; ++i) {
    dh::safe_cuda(hipSetDevice(i));
  }
}

}  // namespace tree
}  // namespace xgboost
