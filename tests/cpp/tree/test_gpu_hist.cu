/*!
 * Copyright 2017-2018 XGBoost contributors
 */

#include <thrust/device_vector.h>
#include <xgboost/base.h>
#include <random>
#include "../helpers.h"
#include "gtest/gtest.h"

#include "../../../src/data/sparse_page_source.h"
#include "../../../src/gbm/gbtree_model.h"
#include "../../../src/tree/updater_gpu_hist.cu"
#include "../../../src/tree/updater_gpu_common.cuh"
#include "../../../src/common/common.h"

namespace xgboost {
namespace tree {

template <typename GradientSumT>
void BuildGidx(DeviceShard<GradientSumT>* shard, int n_rows, int n_cols,
               bst_float sparsity=0) {
  auto dmat = CreateDMatrix(n_rows, n_cols, sparsity, 3);
  const SparsePage& batch = *(*dmat)->GetRowBatches().begin();

  common::HistCutMatrix cmat;
  cmat.row_ptr = {0, 3, 6, 9, 12, 15, 18, 21, 24};
  cmat.min_val = {0.1f, 0.2f, 0.3f, 0.1f, 0.2f, 0.3f, 0.2f, 0.2f};
  // 24 cut fields, 3 cut fields for each feature (column).
  cmat.cut = {0.30f, 0.67f, 1.64f,
              0.32f, 0.77f, 1.95f,
              0.29f, 0.70f, 1.80f,
              0.32f, 0.75f, 1.85f,
              0.18f, 0.59f, 1.69f,
              0.25f, 0.74f, 2.00f,
              0.26f, 0.74f, 1.98f,
              0.26f, 0.71f, 1.83f};

  shard->InitRowPtrs(batch);
  shard->InitCompressedData(cmat, batch);

  delete dmat;
}

TEST(GpuHist, BuildGidxDense) {
  int const n_rows = 16, n_cols = 8;
  TrainParam param;
  param.max_depth = 1;
  param.n_gpus = 1;
  param.max_leaves = 0;

  DeviceShard<GradientPairPrecise> shard(0, 0, n_rows, param);
  BuildGidx(&shard, n_rows, n_cols);

  std::vector<common::CompressedByteT> h_gidx_buffer;
  h_gidx_buffer = shard.gidx_buffer.AsVector();
  common::CompressedIterator<uint32_t> gidx(h_gidx_buffer.data(), 25);

  ASSERT_EQ(shard.row_stride, n_cols);

  std::vector<uint32_t> solution = {
    0, 3, 8,  9, 14, 17, 20, 21,
    0, 4, 7, 10, 14, 16, 19, 22,
    1, 3, 7, 11, 14, 15, 19, 21,
    2, 3, 7,  9, 13, 16, 20, 22,
    2, 3, 6,  9, 12, 16, 20, 21,
    1, 5, 6, 10, 13, 16, 20, 21,
    2, 5, 8,  9, 13, 17, 19, 22,
    2, 4, 6, 10, 14, 17, 19, 21,
    2, 5, 7,  9, 13, 16, 19, 22,
    0, 3, 8, 10, 12, 16, 19, 22,
    1, 3, 7, 10, 13, 16, 19, 21,
    1, 3, 8, 10, 13, 17, 20, 22,
    2, 4, 6,  9, 14, 15, 19, 22,
    1, 4, 6,  9, 13, 16, 19, 21,
    2, 4, 8, 10, 14, 15, 19, 22,
    1, 4, 7, 10, 14, 16, 19, 21,
  };
  for (size_t i = 0; i < n_rows * n_cols; ++i) {
    ASSERT_EQ(solution[i], gidx[i]);
  }
}

TEST(GpuHist, BuildGidxSparse) {
  int const n_rows = 16, n_cols = 8;
  TrainParam param;
  param.max_depth = 1;
  param.n_gpus = 1;
  param.max_leaves = 0;

  DeviceShard<GradientPairPrecise> shard(0, 0, n_rows, param);
  BuildGidx(&shard, n_rows, n_cols, 0.9f);

  std::vector<common::CompressedByteT> h_gidx_buffer;
  h_gidx_buffer = shard.gidx_buffer.AsVector();
  common::CompressedIterator<uint32_t> gidx(h_gidx_buffer.data(), 25);

  ASSERT_LE(shard.row_stride, 3);

  // row_stride = 3, 16 rows, 48 entries for ELLPack
  std::vector<uint32_t> solution = {
    15, 24, 24,  0, 24, 24, 24, 24, 24, 24, 24, 24, 20, 24, 24, 24,
    24, 24, 24, 24, 24,  5, 24, 24,  0, 16, 24, 15, 24, 24, 24, 24,
    24,  7, 14, 16,  4, 24, 24, 24, 24, 24,  9, 24, 24,  1, 24, 24
  };
  for (size_t i = 0; i < n_rows * shard.row_stride; ++i) {
    ASSERT_EQ(solution[i], gidx[i]);
  }
}

std::vector<GradientPairPrecise> GetHostHistGpair() {
  // 24 bins, 3 bins for each feature (column).
  std::vector<GradientPairPrecise> hist_gpair = {
    {0.8314f, 0.7147f}, {1.7989f, 3.7312f}, {3.3846f, 3.4598f},
    {2.9277f, 3.5886f}, {1.8429f, 2.4152f}, {1.2443f, 1.9019f},
    {1.6380f, 2.9174f}, {1.5657f, 2.5107f}, {2.8111f, 2.4776f},
    {2.1322f, 3.0651f}, {3.2927f, 3.8540f}, {0.5899f, 0.9866f},
    {1.5185f, 1.6263f}, {2.0686f, 3.1844f}, {2.4278f, 3.0950f},
    {1.5105f, 2.1403f}, {2.6922f, 4.2217f}, {1.8122f, 1.5437f},
    {0.0000f, 0.0000f}, {4.3245f, 5.7955f}, {1.6903f, 2.1103f},
    {2.4012f, 4.4754f}, {3.6136f, 3.4303f}, {0.0000f, 0.0000f}
  };
  return hist_gpair;
}

template <typename GradientSumT>
void TestBuildHist(GPUHistBuilderBase<GradientSumT>& builder) {
  int const n_rows = 16, n_cols = 8;

  TrainParam param;
  param.max_depth = 6;
  param.n_gpus = 1;
  param.max_leaves = 0;

  DeviceShard<GradientSumT> shard(0, 0, n_rows, param);

  BuildGidx(&shard, n_rows, n_cols);

  xgboost::SimpleLCG gen;
  xgboost::SimpleRealUniformDistribution<bst_float> dist(0.0f, 1.0f);
  std::vector<GradientPair> h_gpair(n_rows);
  for (size_t i = 0; i < h_gpair.size(); ++i) {
    bst_float grad = dist(&gen);
    bst_float hess = dist(&gen);
    h_gpair[i] = GradientPair(grad, hess);
  }

  thrust::device_vector<GradientPair> gpair (n_rows);
  gpair = h_gpair;

  int num_symbols = shard.n_bins + 1;

  thrust::host_vector<common::CompressedByteT> h_gidx_buffer (
      shard.gidx_buffer.Size());

  common::CompressedByteT* d_gidx_buffer_ptr = shard.gidx_buffer.Data();
  dh::safe_cuda(hipMemcpy(h_gidx_buffer.data(), d_gidx_buffer_ptr,
                           sizeof(common::CompressedByteT) * shard.gidx_buffer.Size(),
                           hipMemcpyDeviceToHost));
  auto gidx = common::CompressedIterator<uint32_t>(h_gidx_buffer.data(),
                                                   num_symbols);

  shard.ridx_segments.resize(1);
  shard.ridx_segments[0] = Segment(0, n_rows);
  shard.hist.AllocateHistogram(0);
  shard.gpair.copy(gpair.begin(), gpair.end());
  thrust::sequence(shard.ridx.CurrentDVec().tbegin(),
                   shard.ridx.CurrentDVec().tend());

  builder.Build(&shard, 0);
  DeviceHistogram<GradientSumT> d_hist = shard.hist;

  auto node_histogram = d_hist.GetNodeHistogram(0);
  // d_hist.data stored in float, not gradient pair
  thrust::host_vector<GradientSumT> h_result (d_hist.data.size()/2);
  size_t data_size =
      sizeof(GradientSumT) /
      (sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT));
  data_size *= d_hist.data.size();
  dh::safe_cuda(hipMemcpy(h_result.data(), node_histogram.data(), data_size,
                           hipMemcpyDeviceToHost));

  std::vector<GradientPairPrecise> solution = GetHostHistGpair();
  std::cout << std::fixed;
  for (size_t i = 0; i < h_result.size(); ++i) {
    EXPECT_NEAR(h_result[i].GetGrad(), solution[i].GetGrad(), 0.01f);
    EXPECT_NEAR(h_result[i].GetHess(), solution[i].GetHess(), 0.01f);
  }
}

TEST(GpuHist, BuildHistGlobalMem) {
  GlobalMemHistBuilder<GradientPairPrecise> double_builder;
  TestBuildHist(double_builder);
  GlobalMemHistBuilder<GradientPair> float_builder;
  TestBuildHist(float_builder);
}

TEST(GpuHist, BuildHistSharedMem) {
  SharedMemHistBuilder<GradientPairPrecise> double_builder;
  TestBuildHist(double_builder);
  SharedMemHistBuilder<GradientPair> float_builder;
  TestBuildHist(float_builder);
}

common::HistCutMatrix GetHostCutMatrix () {
  common::HistCutMatrix cmat;
  cmat.row_ptr = {0, 3, 6, 9, 12, 15, 18, 21, 24};
  cmat.min_val = {0.1f, 0.2f, 0.3f, 0.1f, 0.2f, 0.3f, 0.2f, 0.2f};
  // 24 cut fields, 3 cut fields for each feature (column).
  // Each row of the cut represents the cuts for a data column.
  cmat.cut = {0.30f, 0.67f, 1.64f,
              0.32f, 0.77f, 1.95f,
              0.29f, 0.70f, 1.80f,
              0.32f, 0.75f, 1.85f,
              0.18f, 0.59f, 1.69f,
              0.25f, 0.74f, 2.00f,
              0.26f, 0.74f, 1.98f,
              0.26f, 0.71f, 1.83f};
  return cmat;
}

// TODO(trivialfis): This test is over simplified.
TEST(GpuHist, EvaluateSplits) {
  constexpr int n_rows = 16;
  constexpr int n_cols = 8;

  TrainParam param;
  param.max_depth = 1;
  param.n_gpus = 1;
  param.colsample_bynode = 1;
  param.colsample_bylevel = 1;
  param.colsample_bytree = 1;
  param.min_child_weight = 0.01;

  // Disable all parameters.
  param.reg_alpha = 0.0;
  param.reg_lambda = 0;
  param.max_delta_step = 0.0;

  for (size_t i = 0; i < n_cols; ++i) {
    param.monotone_constraints.emplace_back(0);
  }

  int max_bins = 4;

  // Initialize DeviceShard
  std::unique_ptr<DeviceShard<GradientPairPrecise>> shard {new DeviceShard<GradientPairPrecise>(0, 0, n_rows, param)};
  // Initialize DeviceShard::node_sum_gradients
  shard->node_sum_gradients = {{6.4f, 12.8f}};

  // Initialize DeviceShard::cut
  common::HistCutMatrix cmat = GetHostCutMatrix();

  // Copy cut matrix to device.
  DeviceShard<GradientPairPrecise>::DeviceHistCutMatrix cut;
  shard->ba.Allocate(0,
                     &(shard->cut_.feature_segments), cmat.row_ptr.size(),
                     &(shard->cut_.min_fvalue), cmat.min_val.size(),
                     &(shard->cut_.gidx_fvalue_map), 24,
                     &(shard->monotone_constraints), n_cols);
  shard->cut_.feature_segments.copy(cmat.row_ptr.begin(), cmat.row_ptr.end());
  shard->cut_.gidx_fvalue_map.copy(cmat.cut.begin(), cmat.cut.end());
  shard->monotone_constraints.copy(param.monotone_constraints.begin(),
                                   param.monotone_constraints.end());

  // Initialize DeviceShard::hist
  shard->hist.Init(0, (max_bins - 1) * n_cols);
  shard->hist.AllocateHistogram(0);
  // Each row of hist_gpair represents gpairs for one feature.
  // Each entry represents a bin.
  std::vector<GradientPairPrecise> hist_gpair = GetHostHistGpair();
  std::vector<bst_float> hist;
  for (auto pair : hist_gpair) {
    hist.push_back(pair.GetGrad());
    hist.push_back(pair.GetHess());
  }

  ASSERT_EQ(shard->hist.data.size(), hist.size());
  thrust::copy(hist.begin(), hist.end(),
               shard->hist.data.begin());

  // Initialize GPUHistMaker
  GPUHistMakerSpecialised<GradientPairPrecise> hist_maker =
      GPUHistMakerSpecialised<GradientPairPrecise>();
  hist_maker.param_ = param;
  hist_maker.shards_.push_back(std::move(shard));
  hist_maker.column_sampler_.Init(n_cols,
                                  param.colsample_bynode,
                                  param.colsample_bylevel,
                                  param.colsample_bytree,
                                  false);

  RegTree tree;
  MetaInfo info;
  info.num_row_ = n_rows;
  info.num_col_ = n_cols;

  hist_maker.info_ = &info;
  hist_maker.node_value_constraints_.resize(1);
  hist_maker.node_value_constraints_[0].lower_bound = -1.0;
  hist_maker.node_value_constraints_[0].upper_bound = 1.0;

  DeviceSplitCandidate res =
      hist_maker.EvaluateSplit(0, &tree);

  ASSERT_EQ(res.findex, 7);
  ASSERT_NEAR(res.fvalue, 0.26, xgboost::kRtEps);
}

TEST(GpuHist, ApplySplit) {
  GPUHistMakerSpecialised<GradientPairPrecise> hist_maker =
      GPUHistMakerSpecialised<GradientPairPrecise>();
  int constexpr nid = 0;
  int constexpr n_rows = 16;
  int constexpr n_cols = 8;

  TrainParam param;

  // Initialize shard
  for (size_t i = 0; i < n_cols; ++i) {
    param.monotone_constraints.emplace_back(0);
  }

  hist_maker.shards_.resize(1);
  hist_maker.shards_[0].reset(new DeviceShard<GradientPairPrecise>(0, 0, n_rows, param));

  auto& shard = hist_maker.shards_.at(0);
  shard->ridx_segments.resize(3);  // 3 nodes.
  shard->node_sum_gradients.resize(3);

  shard->ridx_segments[0] = Segment(0, n_rows);
  shard->ba.Allocate(0, &(shard->ridx), n_rows,
                     &(shard->position), n_rows);
  shard->row_stride = n_cols;
  thrust::sequence(shard->ridx.CurrentDVec().tbegin(),
                   shard->ridx.CurrentDVec().tend());
  // Initialize GPUHistMaker
  hist_maker.param_ = param;
  RegTree tree;

  DeviceSplitCandidate candidate;
  candidate.Update(2, kLeftDir,
                   0.59, 4,  // fvalue has to be equal to one of the cut field
                   GradientPair(8.2, 2.8), GradientPair(6.3, 3.6),
                   GPUTrainingParam(param));
  GPUHistMakerSpecialised<GradientPairPrecise>::ExpandEntry candidate_entry {0, 0, candidate, 0};
  candidate_entry.nid = nid;

  auto const& nodes = tree.GetNodes();
  size_t n_nodes = nodes.size();

  // Used to get bin_id in update position.
  common::HistCutMatrix cmat = GetHostCutMatrix();
  hist_maker.hmat_ = cmat;

  MetaInfo info;
  info.num_row_ = n_rows;
  info.num_col_ = n_cols;
  info.num_nonzero_ = n_rows * n_cols;  // Dense

  // Initialize gidx
  int n_bins = 24;
  int row_stride = n_cols;
  int num_symbols = n_bins + 1;
  size_t compressed_size_bytes =
      common::CompressedBufferWriter::CalculateBufferSize(
          row_stride * n_rows, num_symbols);
  shard->ba.Allocate(0, &(shard->gidx_buffer), compressed_size_bytes);

  common::CompressedBufferWriter wr(num_symbols);
  std::vector<int> h_gidx (n_rows * row_stride);
  std::iota(h_gidx.begin(), h_gidx.end(), 0);
  std::vector<common::CompressedByteT> h_gidx_compressed (compressed_size_bytes);

  wr.Write(h_gidx_compressed.data(), h_gidx.begin(), h_gidx.end());
  shard->gidx_buffer.copy(h_gidx_compressed.begin(), h_gidx_compressed.end());

  shard->gidx = common::CompressedIterator<uint32_t>(
      shard->gidx_buffer.Data(), num_symbols);

  hist_maker.info_ = &info;
  hist_maker.ApplySplit(candidate_entry, &tree);
  hist_maker.UpdatePosition(candidate_entry, &tree);

  ASSERT_FALSE(tree[nid].IsLeaf());

  int left_nidx = tree[nid].LeftChild();
  int right_nidx = tree[nid].RightChild();

  ASSERT_EQ(shard->ridx_segments[left_nidx].begin, 0);
  ASSERT_EQ(shard->ridx_segments[left_nidx].end, 6);
  ASSERT_EQ(shard->ridx_segments[right_nidx].begin, 6);
  ASSERT_EQ(shard->ridx_segments[right_nidx].end, 16);
}

void TestSortPosition(const std::vector<int>& position_in, int left_idx,
                      int right_idx) {
  int left_count = std::count(position_in.begin(), position_in.end(), left_idx);
  thrust::device_vector<int> position = position_in;
  thrust::device_vector<int> position_out(position.size());

  thrust::device_vector<bst_uint> ridx(position.size());
  thrust::sequence(ridx.begin(), ridx.end());
  thrust::device_vector<bst_uint> ridx_out(ridx.size());
  dh::CubMemory tmp;
  SortPosition(
      &tmp, common::Span<int>(position.data().get(), position.size()),
      common::Span<int>(position_out.data().get(), position_out.size()),
      common::Span<bst_uint>(ridx.data().get(), ridx.size()),
      common::Span<bst_uint>(ridx_out.data().get(), ridx_out.size()), left_idx,
      right_idx, left_count);
  thrust::host_vector<int> position_result = position_out;
  thrust::host_vector<int> ridx_result = ridx_out;

  // Check position is sorted
  EXPECT_TRUE(std::is_sorted(position_result.begin(), position_result.end()));
  // Check row indices are sorted inside left and right segment
  EXPECT_TRUE(
      std::is_sorted(ridx_result.begin(), ridx_result.begin() + left_count));
  EXPECT_TRUE(
      std::is_sorted(ridx_result.begin() + left_count, ridx_result.end()));

  // Check key value pairs are the same
  for (auto i = 0ull; i < ridx_result.size(); i++) {
    EXPECT_EQ(position_result[i], position_in[ridx_result[i]]);
  }
}

TEST(GpuHist, SortPosition) {
  TestSortPosition({1, 2, 1, 2, 1}, 1, 2);
  TestSortPosition({1, 1, 1, 1}, 1, 2);
  TestSortPosition({2, 2, 2, 2}, 1, 2);
  TestSortPosition({1, 2, 1, 2, 3}, 1, 2);
}
}  // namespace tree
}  // namespace xgboost
