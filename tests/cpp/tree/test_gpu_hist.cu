#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2019 XGBoost contributors
 */
#include <thrust/device_vector.h>
#include <xgboost/base.h>
#include <random>
#include <string>
#include <vector>

#include "../helpers.h"
#include "gtest/gtest.h"

#include "../../../src/data/sparse_page_source.h"
#include "../../../src/gbm/gbtree_model.h"
#include "../../../src/tree/updater_gpu_hist.cu"
#include "../../../src/tree/updater_gpu_common.cuh"
#include "../../../src/common/common.h"
#include "../../../src/tree/constraints.cuh"

namespace xgboost {
namespace tree {

TEST(GpuHist, DeviceHistogram) {
  // Ensures that node allocates correctly after reaching `kStopGrowingSize`.
  dh::SaveCudaContext{
    [&]() {
      dh::safe_cuda(hipSetDevice(0));
      constexpr size_t kNBins = 128;
      constexpr size_t kNNodes = 4;
      constexpr size_t kStopGrowing = kNNodes * kNBins * 2u;
      DeviceHistogram<GradientPairPrecise, kStopGrowing> histogram;
      histogram.Init(0, kNBins);
      for (size_t i = 0; i < kNNodes; ++i) {
        histogram.AllocateHistogram(i);
      }
      histogram.Reset();
      ASSERT_EQ(histogram.Data().size(), kStopGrowing);

      // Use allocated memory but do not erase nidx_map.
      for (size_t i = 0; i < kNNodes; ++i) {
        histogram.AllocateHistogram(i);
      }
      for (size_t i = 0; i < kNNodes; ++i) {
        ASSERT_TRUE(histogram.HistogramExists(i));
      }

      // Erase existing nidx_map.
      for (size_t i = kNNodes; i < kNNodes * 2; ++i) {
        histogram.AllocateHistogram(i);
      }
      for (size_t i = 0; i < kNNodes; ++i) {
        ASSERT_FALSE(histogram.HistogramExists(i));
      }
    }
  };
}

namespace {
class HistogramCutsWrapper : public common::HistogramCuts {
 public:
  using SuperT = common::HistogramCuts;
  void SetValues(std::vector<float> cuts) {
    SuperT::cut_values_ = cuts;
  }
  void SetPtrs(std::vector<uint32_t> ptrs) {
    SuperT::cut_ptrs_ = ptrs;
  }
  void SetMins(std::vector<float> mins) {
    SuperT::min_vals_ = mins;
  }
};
}  //  anonymous namespace


template <typename GradientSumT>
void BuildGidx(DeviceShard<GradientSumT>* shard, int n_rows, int n_cols,
               bst_float sparsity=0) {
  auto dmat = CreateDMatrix(n_rows, n_cols, sparsity, 3);
  const SparsePage& batch = *(*dmat)->GetBatches<xgboost::SparsePage>().begin();

  HistogramCutsWrapper cmat;
  cmat.SetPtrs({0, 3, 6, 9, 12, 15, 18, 21, 24});
  // 24 cut fields, 3 cut fields for each feature (column).
  cmat.SetValues({0.30f, 0.67f, 1.64f,
          0.32f, 0.77f, 1.95f,
          0.29f, 0.70f, 1.80f,
          0.32f, 0.75f, 1.85f,
          0.18f, 0.59f, 1.69f,
          0.25f, 0.74f, 2.00f,
          0.26f, 0.74f, 1.98f,
          0.26f, 0.71f, 1.83f});
  cmat.SetMins({0.1f, 0.2f, 0.3f, 0.1f, 0.2f, 0.3f, 0.2f, 0.2f});

  auto is_dense = (*dmat)->Info().num_nonzero_ ==
                  (*dmat)->Info().num_row_ * (*dmat)->Info().num_col_;
  size_t row_stride = 0;
  const auto &offset_vec = batch.offset.ConstHostVector();
  for (size_t i = 1; i < offset_vec.size(); ++i) {
    row_stride = std::max(row_stride, offset_vec[i] - offset_vec[i-1]);
  }
  shard->InitCompressedData(cmat, row_stride, is_dense);
  shard->CreateHistIndices(
    batch, cmat, RowStateOnDevice(batch.Size(), batch.Size()), -1);

  delete dmat;
}

TEST(GpuHist, BuildGidxDense) {
  int constexpr kNRows = 16, kNCols = 8;
  tree::TrainParam param;
  std::vector<std::pair<std::string, std::string>> args {
    {"max_depth", "1"},
    {"max_leaves", "0"},
  };
  param.Init(args);
  DeviceShard<GradientPairPrecise> shard(0, 0, 0, kNRows, param, kNCols, kNCols);
  BuildGidx(&shard, kNRows, kNCols);

  std::vector<common::CompressedByteT> h_gidx_buffer(shard.gidx_buffer.size());
  dh::CopyDeviceSpanToVector(&h_gidx_buffer, shard.gidx_buffer);
  common::CompressedIterator<uint32_t> gidx(h_gidx_buffer.data(), 25);

  ASSERT_EQ(shard.ellpack_matrix.row_stride, kNCols);

  std::vector<uint32_t> solution = {
    0, 3, 8,  9, 14, 17, 20, 21,
    0, 4, 7, 10, 14, 16, 19, 22,
    1, 3, 7, 11, 14, 15, 19, 21,
    2, 3, 7,  9, 13, 16, 20, 22,
    2, 3, 6,  9, 12, 16, 20, 21,
    1, 5, 6, 10, 13, 16, 20, 21,
    2, 5, 8,  9, 13, 17, 19, 22,
    2, 4, 6, 10, 14, 17, 19, 21,
    2, 5, 7,  9, 13, 16, 19, 22,
    0, 3, 8, 10, 12, 16, 19, 22,
    1, 3, 7, 10, 13, 16, 19, 21,
    1, 3, 8, 10, 13, 17, 20, 22,
    2, 4, 6,  9, 14, 15, 19, 22,
    1, 4, 6,  9, 13, 16, 19, 21,
    2, 4, 8, 10, 14, 15, 19, 22,
    1, 4, 7, 10, 14, 16, 19, 21,
  };
  for (size_t i = 0; i < kNRows * kNCols; ++i) {
    ASSERT_EQ(solution[i], gidx[i]);
  }
}

TEST(GpuHist, BuildGidxSparse) {
  int constexpr kNRows = 16, kNCols = 8;
  TrainParam param;
  std::vector<std::pair<std::string, std::string>> args {
    {"max_depth", "1"},
    {"max_leaves", "0"},
  };
  param.Init(args);

  DeviceShard<GradientPairPrecise> shard(0, 0, 0, kNRows, param, kNCols,
                                         kNCols);
  BuildGidx(&shard, kNRows, kNCols, 0.9f);

  std::vector<common::CompressedByteT> h_gidx_buffer(shard.gidx_buffer.size());
  dh::CopyDeviceSpanToVector(&h_gidx_buffer, shard.gidx_buffer);
  common::CompressedIterator<uint32_t> gidx(h_gidx_buffer.data(), 25);

  ASSERT_LE(shard.ellpack_matrix.row_stride, 3);

  // row_stride = 3, 16 rows, 48 entries for ELLPack
  std::vector<uint32_t> solution = {
    15, 24, 24,  0, 24, 24, 24, 24, 24, 24, 24, 24, 20, 24, 24, 24,
    24, 24, 24, 24, 24,  5, 24, 24,  0, 16, 24, 15, 24, 24, 24, 24,
    24,  7, 14, 16,  4, 24, 24, 24, 24, 24,  9, 24, 24,  1, 24, 24
  };
  for (size_t i = 0; i < kNRows * shard.ellpack_matrix.row_stride; ++i) {
    ASSERT_EQ(solution[i], gidx[i]);
  }
}

std::vector<GradientPairPrecise> GetHostHistGpair() {
  // 24 bins, 3 bins for each feature (column).
  std::vector<GradientPairPrecise> hist_gpair = {
    {0.8314f, 0.7147f}, {1.7989f, 3.7312f}, {3.3846f, 3.4598f},
    {2.9277f, 3.5886f}, {1.8429f, 2.4152f}, {1.2443f, 1.9019f},
    {1.6380f, 2.9174f}, {1.5657f, 2.5107f}, {2.8111f, 2.4776f},
    {2.1322f, 3.0651f}, {3.2927f, 3.8540f}, {0.5899f, 0.9866f},
    {1.5185f, 1.6263f}, {2.0686f, 3.1844f}, {2.4278f, 3.0950f},
    {1.5105f, 2.1403f}, {2.6922f, 4.2217f}, {1.8122f, 1.5437f},
    {0.0000f, 0.0000f}, {4.3245f, 5.7955f}, {1.6903f, 2.1103f},
    {2.4012f, 4.4754f}, {3.6136f, 3.4303f}, {0.0000f, 0.0000f}
  };
  return hist_gpair;
}

template <typename GradientSumT>
void TestBuildHist(bool use_shared_memory_histograms) {
  int const kNRows = 16, kNCols = 8;

  TrainParam param;
  std::vector<std::pair<std::string, std::string>> args {
    {"max_depth", "6"},
    {"max_leaves", "0"},
  };
  param.Init(args);
  DeviceShard<GradientSumT> shard(0, 0, 0, kNRows, param, kNCols,
                                  kNCols);
  BuildGidx(&shard, kNRows, kNCols);

  xgboost::SimpleLCG gen;
  xgboost::SimpleRealUniformDistribution<bst_float> dist(0.0f, 1.0f);
  std::vector<GradientPair> h_gpair(kNRows);
  for (auto &gpair : h_gpair) {
    bst_float grad = dist(&gen);
    bst_float hess = dist(&gen);
    gpair = GradientPair(grad, hess);
  }

  thrust::host_vector<common::CompressedByteT> h_gidx_buffer (
      shard.gidx_buffer.size());

  common::CompressedByteT* d_gidx_buffer_ptr = shard.gidx_buffer.data();
  dh::safe_cuda(hipMemcpy(h_gidx_buffer.data(), d_gidx_buffer_ptr,
                           sizeof(common::CompressedByteT) * shard.gidx_buffer.size(),
                           hipMemcpyDeviceToHost));

  shard.row_partitioner.reset(new RowPartitioner(0, kNRows));
  shard.hist.AllocateHistogram(0);
  dh::CopyVectorToDeviceSpan(shard.gpair, h_gpair);

  shard.use_shared_memory_histograms = use_shared_memory_histograms;
  shard.BuildHist(0);
  DeviceHistogram<GradientSumT> d_hist = shard.hist;

  auto node_histogram = d_hist.GetNodeHistogram(0);
  // d_hist.data stored in float, not gradient pair
  thrust::host_vector<GradientSumT> h_result (d_hist.Data().size() / 2);
  size_t data_size =
      sizeof(GradientSumT) /
      (sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT));
  data_size *= d_hist.Data().size();
  dh::safe_cuda(hipMemcpy(h_result.data(), node_histogram.data(), data_size,
                           hipMemcpyDeviceToHost));

  std::vector<GradientPairPrecise> solution = GetHostHistGpair();
  std::cout << std::fixed;
  for (size_t i = 0; i < h_result.size(); ++i) {
    EXPECT_NEAR(h_result[i].GetGrad(), solution[i].GetGrad(), 0.01f);
    EXPECT_NEAR(h_result[i].GetHess(), solution[i].GetHess(), 0.01f);
  }
}

TEST(GpuHist, BuildHistGlobalMem) {
  TestBuildHist<GradientPairPrecise>(false);
  TestBuildHist<GradientPair>(false);
}

TEST(GpuHist, BuildHistSharedMem) {
  TestBuildHist<GradientPairPrecise>(true);
  TestBuildHist<GradientPair>(true);
}

HistogramCutsWrapper GetHostCutMatrix () {
  HistogramCutsWrapper cmat;
  cmat.SetPtrs({0, 3, 6, 9, 12, 15, 18, 21, 24});
  cmat.SetMins({0.1f, 0.2f, 0.3f, 0.1f, 0.2f, 0.3f, 0.2f, 0.2f});
  // 24 cut fields, 3 cut fields for each feature (column).
  // Each row of the cut represents the cuts for a data column.
  cmat.SetValues({0.30f, 0.67f, 1.64f,
              0.32f, 0.77f, 1.95f,
              0.29f, 0.70f, 1.80f,
              0.32f, 0.75f, 1.85f,
              0.18f, 0.59f, 1.69f,
              0.25f, 0.74f, 2.00f,
              0.26f, 0.74f, 1.98f,
              0.26f, 0.71f, 1.83f});
  return cmat;
}

// TODO(trivialfis): This test is over simplified.
TEST(GpuHist, EvaluateSplits) {
  constexpr int kNRows = 16;
  constexpr int kNCols = 8;

  TrainParam param;

  std::vector<std::pair<std::string, std::string>> args {
    {"max_depth", "1"},
    {"max_leaves", "0"},

    // Disable all other parameters.
    {"colsample_bynode", "1"},
    {"colsample_bylevel", "1"},
    {"colsample_bytree", "1"},
    {"min_child_weight", "0.01"},
    {"reg_alpha", "0"},
    {"reg_lambda", "0"},
    {"max_delta_step", "0"}
  };
  param.Init(args);
  for (size_t i = 0; i < kNCols; ++i) {
    param.monotone_constraints.emplace_back(0);
  }

  int max_bins = 4;

  // Initialize DeviceShard
  std::unique_ptr<DeviceShard<GradientPairPrecise>> shard{
    new DeviceShard<GradientPairPrecise>(0, 0, 0, kNRows, param, kNCols,
                                         kNCols)};
  // Initialize DeviceShard::node_sum_gradients
  shard->node_sum_gradients = {{6.4f, 12.8f}};

  // Initialize DeviceShard::cut
  auto cmat = GetHostCutMatrix();

  // Copy cut matrix to device.
  shard->ba.Allocate(0,
                     &(shard->feature_segments), cmat.Ptrs().size(),
                     &(shard->min_fvalue), cmat.MinValues().size(),
                     &(shard->gidx_fvalue_map), 24,
                     &(shard->monotone_constraints), kNCols);
  dh::CopyVectorToDeviceSpan(shard->feature_segments, cmat.Ptrs());
  dh::CopyVectorToDeviceSpan(shard->gidx_fvalue_map, cmat.Values());
  dh::CopyVectorToDeviceSpan(shard->monotone_constraints,
                             param.monotone_constraints);
  shard->ellpack_matrix.feature_segments = shard->feature_segments;
  shard->ellpack_matrix.gidx_fvalue_map = shard->gidx_fvalue_map;
  dh::CopyVectorToDeviceSpan(shard->min_fvalue, cmat.MinValues());
  shard->ellpack_matrix.min_fvalue = shard->min_fvalue;

  // Initialize DeviceShard::hist
  shard->hist.Init(0, (max_bins - 1) * kNCols);
  shard->hist.AllocateHistogram(0);
  // Each row of hist_gpair represents gpairs for one feature.
  // Each entry represents a bin.
  std::vector<GradientPairPrecise> hist_gpair = GetHostHistGpair();
  std::vector<bst_float> hist;
  for (auto pair : hist_gpair) {
    hist.push_back(pair.GetGrad());
    hist.push_back(pair.GetHess());
  }

  ASSERT_EQ(shard->hist.Data().size(), hist.size());
  thrust::copy(hist.begin(), hist.end(),
               shard->hist.Data().begin());

  shard->column_sampler.Init(kNCols,
                                  param.colsample_bynode,
                                  param.colsample_bylevel,
                                  param.colsample_bytree,
                                  false);

  RegTree tree;
  MetaInfo info;
  info.num_row_ = kNRows;
  info.num_col_ = kNCols;

  shard->node_value_constraints.resize(1);
  shard->node_value_constraints[0].lower_bound = -1.0;
  shard->node_value_constraints[0].upper_bound = 1.0;

  std::vector<DeviceSplitCandidate> res =
    shard->EvaluateSplits({ 0,0 }, tree, kNCols);

  ASSERT_EQ(res[0].findex, 7);
  ASSERT_EQ(res[1].findex, 7);
  ASSERT_NEAR(res[0].fvalue, 0.26, xgboost::kRtEps);
  ASSERT_NEAR(res[1].fvalue, 0.26, xgboost::kRtEps);
}

void TestHistogramIndexImpl(int n_gpus) {
  // Test if the compressed histogram index matches when using a sparse
  // dmatrix with and without using external memory

  int constexpr kNRows = 1000, kNCols = 10;

  // Build 2 matrices and build a histogram maker with that
  tree::GPUHistMakerSpecialised<GradientPairPrecise> hist_maker, hist_maker_ext;
  std::unique_ptr<DMatrix> hist_maker_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 0, true));
  std::unique_ptr<DMatrix> hist_maker_ext_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 128UL, true));

  std::vector<std::pair<std::string, std::string>> training_params = {
    {"max_depth", "10"},
    {"max_leaves", "0"}
  };

  GenericParameter generic_param(CreateEmptyGenericParam(0, n_gpus));
  hist_maker.Configure(training_params, &generic_param);

  hist_maker.InitDataOnce(hist_maker_dmat.get());
  hist_maker_ext.Configure(training_params, &generic_param);
  hist_maker_ext.InitDataOnce(hist_maker_ext_dmat.get());

  ASSERT_EQ(hist_maker.shards_.size(), hist_maker_ext.shards_.size());

  // Extract the device shards from the histogram makers and from that its compressed
  // histogram index
  for (size_t i = 0; i < hist_maker.shards_.size(); ++i) {
    const auto &dev_shard = hist_maker.shards_[i];
    std::vector<common::CompressedByteT> h_gidx_buffer(dev_shard->gidx_buffer.size());
    dh::CopyDeviceSpanToVector(&h_gidx_buffer, dev_shard->gidx_buffer);

    const auto &dev_shard_ext = hist_maker_ext.shards_[i];
    std::vector<common::CompressedByteT> h_gidx_buffer_ext(dev_shard_ext->gidx_buffer.size());
    dh::CopyDeviceSpanToVector(&h_gidx_buffer_ext, dev_shard_ext->gidx_buffer);

    ASSERT_EQ(dev_shard->n_bins, dev_shard_ext->n_bins);
    ASSERT_EQ(dev_shard->gidx_buffer.size(), dev_shard_ext->gidx_buffer.size());

    ASSERT_EQ(h_gidx_buffer, h_gidx_buffer_ext);
  }
}

TEST(GpuHist, TestHistogramIndex) {
  TestHistogramIndexImpl(1);
}

}  // namespace tree
}  // namespace xgboost
