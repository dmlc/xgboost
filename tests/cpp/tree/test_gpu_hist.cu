/*!
 * Copyright 2017-2022 XGBoost contributors
 */
#include <gtest/gtest.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <xgboost/base.h>

#include <random>
#include <string>
#include <vector>

#include "../../../src/common/common.h"
#include "../../../src/data/sparse_page_source.h"
#include "../../../src/tree/constraints.cuh"
#include "../../../src/tree/updater_gpu_common.cuh"
#include "../../../src/tree/updater_gpu_hist.cu"
#include "../filesystem.h"  // dmlc::TemporaryDirectory
#include "../helpers.h"
#include "../histogram_helpers.h"
#include "xgboost/context.h"
#include "xgboost/json.h"

namespace xgboost {
namespace tree {
TEST(GpuHist, DeviceHistogram) {
  // Ensures that node allocates correctly after reaching `kStopGrowingSize`.
  dh::safe_cuda(hipSetDevice(0));
  constexpr size_t kNBins = 128;
  constexpr int kNNodes = 4;
  constexpr size_t kStopGrowing = kNNodes * kNBins * 2u;
  DeviceHistogramStorage<kStopGrowing> histogram;
  histogram.Init(0, kNBins);
  for (int i = 0; i < kNNodes; ++i) {
    histogram.AllocateHistograms({i});
  }
  histogram.Reset();
  ASSERT_EQ(histogram.Data().size(), kStopGrowing);

  // Use allocated memory but do not erase nidx_map.
  for (int i = 0; i < kNNodes; ++i) {
    histogram.AllocateHistograms({i});
  }
  for (int i = 0; i < kNNodes; ++i) {
    ASSERT_TRUE(histogram.HistogramExists(i));
  }

  // Add two new nodes
  histogram.AllocateHistograms({kNNodes});
  histogram.AllocateHistograms({kNNodes + 1});

  // Old cached nodes should still exist
  for (int i = 0; i < kNNodes; ++i) {
    ASSERT_TRUE(histogram.HistogramExists(i));
  }

  // Should be deleted
  ASSERT_FALSE(histogram.HistogramExists(kNNodes));
  // Most recent node should exist
  ASSERT_TRUE(histogram.HistogramExists(kNNodes + 1));

  // Add same node again - should fail
  EXPECT_ANY_THROW(histogram.AllocateHistograms({kNNodes + 1}););
}

std::vector<GradientPairPrecise> GetHostHistGpair() {
  // 24 bins, 3 bins for each feature (column).
  std::vector<GradientPairPrecise> hist_gpair = {
    {0.8314f, 0.7147f}, {1.7989f, 3.7312f}, {3.3846f, 3.4598f},
    {2.9277f, 3.5886f}, {1.8429f, 2.4152f}, {1.2443f, 1.9019f},
    {1.6380f, 2.9174f}, {1.5657f, 2.5107f}, {2.8111f, 2.4776f},
    {2.1322f, 3.0651f}, {3.2927f, 3.8540f}, {0.5899f, 0.9866f},
    {1.5185f, 1.6263f}, {2.0686f, 3.1844f}, {2.4278f, 3.0950f},
    {1.5105f, 2.1403f}, {2.6922f, 4.2217f}, {1.8122f, 1.5437f},
    {0.0000f, 0.0000f}, {4.3245f, 5.7955f}, {1.6903f, 2.1103f},
    {2.4012f, 4.4754f}, {3.6136f, 3.4303f}, {0.0000f, 0.0000f}
  };
  return hist_gpair;
}

template <typename GradientSumT>
void TestBuildHist(bool use_shared_memory_histograms) {
  int const kNRows = 16, kNCols = 8;

  TrainParam param;
  std::vector<std::pair<std::string, std::string>> args {
    {"max_depth", "6"},
    {"max_leaves", "0"},
  };
  param.Init(args);
  auto page = BuildEllpackPage(kNRows, kNCols);
  BatchParam batch_param{};
  Context ctx{CreateEmptyGenericParam(0)};
  GPUHistMakerDevice<GradientSumT> maker(&ctx, page.get(), {}, kNRows, param, kNCols, kNCols,
                                         batch_param);
  xgboost::SimpleLCG gen;
  xgboost::SimpleRealUniformDistribution<bst_float> dist(0.0f, 1.0f);
  HostDeviceVector<GradientPair> gpair(kNRows);
  for (auto &gp : gpair.HostVector()) {
    bst_float grad = dist(&gen);
    bst_float hess = dist(&gen);
    gp = GradientPair(grad, hess);
  }
  gpair.SetDevice(0);

  thrust::host_vector<common::CompressedByteT> h_gidx_buffer (page->gidx_buffer.HostVector());
  maker.row_partitioner.reset(new RowPartitioner(0, kNRows));
  maker.hist.AllocateHistograms({0});
  maker.gpair = gpair.DeviceSpan();
  maker.quantiser.reset(new GradientQuantiser(maker.gpair));

  BuildGradientHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(0),
                         maker.feature_groups->DeviceAccessor(0), gpair.DeviceSpan(),
                         maker.row_partitioner->GetRows(0), maker.hist.GetNodeHistogram(0),
                         *maker.quantiser, !use_shared_memory_histograms);

  DeviceHistogramStorage<>& d_hist = maker.hist;

  auto node_histogram = d_hist.GetNodeHistogram(0);
  // d_hist.data stored in float, not gradient pair
  thrust::host_vector<GradientPairInt64> h_result (node_histogram.size());
  dh::safe_cuda(hipMemcpy(h_result.data(), node_histogram.data(), node_histogram.size_bytes(),
                           hipMemcpyDeviceToHost));

  std::vector<GradientPairPrecise> solution = GetHostHistGpair();
  for (size_t i = 0; i < h_result.size(); ++i) {
    auto result = maker.quantiser->ToFloatingPoint(h_result[i]);
    EXPECT_NEAR(result.GetGrad(), solution[i].GetGrad(), 0.01f);
    EXPECT_NEAR(result.GetHess(), solution[i].GetHess(), 0.01f);
  }
}

TEST(GpuHist, BuildHistGlobalMem) {
  TestBuildHist<GradientPairPrecise>(false);
}

TEST(GpuHist, BuildHistSharedMem) {
  TestBuildHist<GradientPairPrecise>(true);
}

HistogramCutsWrapper GetHostCutMatrix () {
  HistogramCutsWrapper cmat;
  cmat.SetPtrs({0, 3, 6, 9, 12, 15, 18, 21, 24});
  cmat.SetMins({0.1f, 0.2f, 0.3f, 0.1f, 0.2f, 0.3f, 0.2f, 0.2f});
  // 24 cut fields, 3 cut fields for each feature (column).
  // Each row of the cut represents the cuts for a data column.
  cmat.SetValues({0.30f, 0.67f, 1.64f,
              0.32f, 0.77f, 1.95f,
              0.29f, 0.70f, 1.80f,
              0.32f, 0.75f, 1.85f,
              0.18f, 0.59f, 1.69f,
              0.25f, 0.74f, 2.00f,
              0.26f, 0.74f, 1.98f,
              0.26f, 0.71f, 1.83f});
  return cmat;
}

inline GradientQuantiser DummyRoundingFactor() {
  thrust::device_vector<GradientPair> gpair(1);
  gpair[0] = {1000.f, 1000.f};  // Tests should not exceed sum of 1000
  return GradientQuantiser(dh::ToSpan(gpair));
}

void TestHistogramIndexImpl() {
  // Test if the compressed histogram index matches when using a sparse
  // dmatrix with and without using external memory

  int constexpr kNRows = 1000, kNCols = 10;

  // Build 2 matrices and build a histogram maker with that

  Context ctx(CreateEmptyGenericParam(0));
  tree::GPUHistMaker hist_maker{&ctx, ObjInfo{ObjInfo::kRegression}},
      hist_maker_ext{&ctx, ObjInfo{ObjInfo::kRegression}};
  std::unique_ptr<DMatrix> hist_maker_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 0, true));

  dmlc::TemporaryDirectory tempdir;
  std::unique_ptr<DMatrix> hist_maker_ext_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 128UL, true, tempdir));

  std::vector<std::pair<std::string, std::string>> training_params = {
    {"max_depth", "10"},
    {"max_leaves", "0"}
  };

  hist_maker.Configure(training_params);
  hist_maker.InitDataOnce(hist_maker_dmat.get());
  hist_maker_ext.Configure(training_params);
  hist_maker_ext.InitDataOnce(hist_maker_ext_dmat.get());

  // Extract the device maker from the histogram makers and from that its compressed
  // histogram index
  const auto &maker = hist_maker.maker;
  auto grad = GenerateRandomGradients(kNRows);
  grad.SetDevice(0);
  maker->Reset(&grad, hist_maker_dmat.get(), kNCols);
  std::vector<common::CompressedByteT> h_gidx_buffer(maker->page->gidx_buffer.HostVector());

  const auto &maker_ext = hist_maker_ext.maker;
  maker_ext->Reset(&grad, hist_maker_ext_dmat.get(), kNCols);
  std::vector<common::CompressedByteT> h_gidx_buffer_ext(maker_ext->page->gidx_buffer.HostVector());

  ASSERT_EQ(maker->page->Cuts().TotalBins(), maker_ext->page->Cuts().TotalBins());
  ASSERT_EQ(maker->page->gidx_buffer.Size(), maker_ext->page->gidx_buffer.Size());
}

TEST(GpuHist, TestHistogramIndex) {
  TestHistogramIndexImpl();
}

void UpdateTree(HostDeviceVector<GradientPair>* gpair, DMatrix* dmat,
                size_t gpu_page_size, RegTree* tree,
                HostDeviceVector<bst_float>* preds, float subsample = 1.0f,
                const std::string& sampling_method = "uniform",
                int max_bin = 2) {

  if (gpu_page_size > 0) {
    // Loop over the batches and count the records
    int64_t batch_count = 0;
    int64_t row_count = 0;
    for (const auto& batch : dmat->GetBatches<EllpackPage>({0, max_bin})) {
      EXPECT_LT(batch.Size(), dmat->Info().num_row_);
      batch_count++;
      row_count += batch.Size();
    }
    EXPECT_GE(batch_count, 2);
    EXPECT_EQ(row_count, dmat->Info().num_row_);
  }

  Args args{
      {"max_depth", "2"},
      {"max_bin", std::to_string(max_bin)},
      {"min_child_weight", "0.0"},
      {"reg_alpha", "0"},
      {"reg_lambda", "0"},
      {"subsample", std::to_string(subsample)},
      {"sampling_method", sampling_method},
  };

  Context ctx(CreateEmptyGenericParam(0));
  tree::GPUHistMaker hist_maker{&ctx,ObjInfo{ObjInfo::kRegression}};
  hist_maker.Configure(args);

  std::vector<HostDeviceVector<bst_node_t>> position(1);
  hist_maker.Update(gpair, dmat, common::Span<HostDeviceVector<bst_node_t>>{position}, {tree});
  auto cache = linalg::VectorView<float>{preds->DeviceSpan(), {preds->Size()}, 0};
  hist_maker.UpdatePredictionCache(dmat, cache);
}

TEST(GpuHist, UniformSampling) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr float kSubsample = 0.9999;
  common::GlobalRandom().seed(1994);

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  auto gpair = GenerateRandomGradients(kRows);

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree, &preds, 1.0, "uniform", kRows);
  // Build another tree using sampling.
  RegTree tree_sampling;
  HostDeviceVector<bst_float> preds_sampling(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree_sampling, &preds_sampling, kSubsample,
             "uniform", kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_sampling_h = preds_sampling.ConstHostVector();
  for (size_t i = 0; i < kRows; i++) {
    EXPECT_NEAR(preds_h[i], preds_sampling_h[i], 1e-8);
  }
}

TEST(GpuHist, GradientBasedSampling) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr float kSubsample = 0.9999;
  common::GlobalRandom().seed(1994);

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  auto gpair = GenerateRandomGradients(kRows);

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree, &preds, 1.0, "uniform", kRows);

  // Build another tree using sampling.
  RegTree tree_sampling;
  HostDeviceVector<bst_float> preds_sampling(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree_sampling, &preds_sampling, kSubsample,
             "gradient_based", kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_sampling_h = preds_sampling.ConstHostVector();
  for (size_t i = 0; i < kRows; i++) {
    EXPECT_NEAR(preds_h[i], preds_sampling_h[i], 1e-3);
  }
}

TEST(GpuHist, ExternalMemory) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr size_t kPageSize = 1024;

  dmlc::TemporaryDirectory tmpdir;

  // Create a DMatrix with multiple batches.
  std::unique_ptr<DMatrix> dmat_ext(
      CreateSparsePageDMatrix(kRows, kCols, kRows / kPageSize, tmpdir.path + "/cache"));

  // Create a single batch DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrix(kRows, kCols, 1, tmpdir.path + "/cache"));

  auto gpair = GenerateRandomGradients(kRows);

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree, &preds, 1.0, "uniform", kRows);
  // Build another tree using multiple ELLPACK pages.
  RegTree tree_ext;
  HostDeviceVector<bst_float> preds_ext(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat_ext.get(), kPageSize, &tree_ext, &preds_ext, 1.0, "uniform", kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_ext_h = preds_ext.ConstHostVector();
  for (size_t i = 0; i < kRows; i++) {
    EXPECT_NEAR(preds_h[i], preds_ext_h[i], 1e-6);
  }
}

TEST(GpuHist, ExternalMemoryWithSampling) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 2;
  constexpr size_t kPageSize = 1024;
  constexpr float kSubsample = 0.5;
  const std::string kSamplingMethod = "gradient_based";
  common::GlobalRandom().seed(0);

  dmlc::TemporaryDirectory tmpdir;

  // Create a single batch DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrix(kRows, kCols, 1, tmpdir.path + "/cache"));

  // Create a DMatrix with multiple batches.
  std::unique_ptr<DMatrix> dmat_ext(
      CreateSparsePageDMatrix(kRows, kCols, kRows / kPageSize, tmpdir.path + "/cache"));

  auto gpair = GenerateRandomGradients(kRows);

  // Build a tree using the in-memory DMatrix.
  auto rng = common::GlobalRandom();

  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree, &preds, kSubsample, kSamplingMethod,
             kRows);

  // Build another tree using multiple ELLPACK pages.
  common::GlobalRandom() = rng;
  RegTree tree_ext;
  HostDeviceVector<bst_float> preds_ext(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat_ext.get(), kPageSize, &tree_ext, &preds_ext,
             kSubsample, kSamplingMethod, kRows);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_ext_h = preds_ext.ConstHostVector();
  for (size_t i = 0; i < kRows; i++) {
    ASSERT_NEAR(preds_h[i], preds_ext_h[i], 1e-3);
  }
}

TEST(GpuHist, ConfigIO) {
  Context ctx(CreateEmptyGenericParam(0));
  std::unique_ptr<TreeUpdater> updater{
      TreeUpdater::Create("grow_gpu_hist", &ctx, ObjInfo{ObjInfo::kRegression})};
  updater->Configure(Args{});

  Json j_updater { Object() };
  updater->SaveConfig(&j_updater);
  ASSERT_TRUE(IsA<Object>(j_updater["gpu_hist_train_param"]));
  ASSERT_TRUE(IsA<Object>(j_updater["train_param"]));
  updater->LoadConfig(j_updater);

  Json j_updater_roundtrip { Object() };
  updater->SaveConfig(&j_updater_roundtrip);
  ASSERT_TRUE(IsA<Object>(j_updater_roundtrip["gpu_hist_train_param"]));
  ASSERT_TRUE(IsA<Object>(j_updater_roundtrip["train_param"]));

  ASSERT_EQ(j_updater, j_updater_roundtrip);
}

TEST(GpuHist, MaxDepth) {
  Context ctx(CreateEmptyGenericParam(0));
  size_t constexpr kRows = 16;
  size_t constexpr kCols = 4;
  auto p_mat = RandomDataGenerator{kRows, kCols, 0}.GenerateDMatrix();

  auto learner = std::unique_ptr<Learner>(Learner::Create({p_mat}));
  learner->SetParam("max_depth", "32");
  learner->Configure();

  ASSERT_THROW({learner->UpdateOneIter(0, p_mat);}, dmlc::Error);
}
}  // namespace tree
}  // namespace xgboost
