#include "hip/hip_runtime.h"
/*!
 * Copyright 2017-2018 XGBoost contributors
 */

#include <thrust/device_vector.h>
#include <xgboost/base.h>
#include <random>
#include <string>
#include <vector>

#include "../helpers.h"
#include "gtest/gtest.h"

#include "../../../src/data/sparse_page_source.h"
#include "../../../src/gbm/gbtree_model.h"
#include "../../../src/tree/updater_gpu_hist.cu"
#include "../../../src/tree/updater_gpu_common.cuh"
#include "../../../src/common/common.h"

namespace xgboost {
namespace tree {

TEST(GpuHist, DeviceHistogram) {
  // Ensures that node allocates correctly after reaching `kStopGrowingSize`.
  dh::SaveCudaContext{
    [&]() {
      dh::safe_cuda(hipSetDevice(0));
      constexpr size_t kNBins = 128;
      constexpr size_t kNNodes = 4;
      constexpr size_t kStopGrowing = kNNodes * kNBins * 2u;
      DeviceHistogram<GradientPairPrecise, kStopGrowing> histogram;
      histogram.Init(0, kNBins);
      for (size_t i = 0; i < kNNodes; ++i) {
        histogram.AllocateHistogram(i);
      }
      histogram.Reset();
      ASSERT_EQ(histogram.Data().size(), kStopGrowing);

      // Use allocated memory but do not erase nidx_map.
      for (size_t i = 0; i < kNNodes; ++i) {
        histogram.AllocateHistogram(i);
      }
      for (size_t i = 0; i < kNNodes; ++i) {
        ASSERT_TRUE(histogram.HistogramExists(i));
      }

      // Erase existing nidx_map.
      for (size_t i = kNNodes; i < kNNodes * 2; ++i) {
        histogram.AllocateHistogram(i);
      }
      for (size_t i = 0; i < kNNodes; ++i) {
        ASSERT_FALSE(histogram.HistogramExists(i));
      }
    }
  };

}

template <typename GradientSumT>
void BuildGidx(DeviceShard<GradientSumT>* shard, int n_rows, int n_cols,
               bst_float sparsity=0) {
  auto dmat = CreateDMatrix(n_rows, n_cols, sparsity, 3);
  const SparsePage& batch = *(*dmat)->GetRowBatches().begin();

  common::HistCutMatrix cmat;
  cmat.row_ptr = {0, 3, 6, 9, 12, 15, 18, 21, 24};
  cmat.min_val = {0.1f, 0.2f, 0.3f, 0.1f, 0.2f, 0.3f, 0.2f, 0.2f};
  // 24 cut fields, 3 cut fields for each feature (column).
  cmat.cut = {0.30f, 0.67f, 1.64f,
              0.32f, 0.77f, 1.95f,
              0.29f, 0.70f, 1.80f,
              0.32f, 0.75f, 1.85f,
              0.18f, 0.59f, 1.69f,
              0.25f, 0.74f, 2.00f,
              0.26f, 0.74f, 1.98f,
              0.26f, 0.71f, 1.83f};

  auto is_dense = (*dmat)->Info().num_nonzero_ ==
                  (*dmat)->Info().num_row_ * (*dmat)->Info().num_col_;
  size_t row_stride = 0;
  const auto &offset_vec = batch.offset.ConstHostVector();
  for (size_t i = 1; i < offset_vec.size(); ++i) {
    row_stride = std::max(row_stride, offset_vec[i] - offset_vec[i-1]);
  }
  shard->InitCompressedData(cmat, row_stride, is_dense);
  shard->CreateHistIndices(batch, cmat,
                           std::vector<std::pair<size_t, size_t>>(1, {0, batch.Size()}), -1);

  delete dmat;
}

TEST(GpuHist, BuildGidxDense) {
  int constexpr kNRows = 16, kNCols = 8;
  TrainParam param;
  param.max_depth = 1;
  param.n_gpus = 1;
  param.max_leaves = 0;

  DeviceShard<GradientPairPrecise> shard(0, 0, 0, kNRows, param, kNCols);
  BuildGidx(&shard, kNRows, kNCols);

  std::vector<common::CompressedByteT> h_gidx_buffer(shard.gidx_buffer.size());
  dh::CopyDeviceSpanToVector(&h_gidx_buffer, shard.gidx_buffer);
  common::CompressedIterator<uint32_t> gidx(h_gidx_buffer.data(), 25);

  ASSERT_EQ(shard.ellpack_matrix.row_stride, kNCols);

  std::vector<uint32_t> solution = {
    0, 3, 8,  9, 14, 17, 20, 21,
    0, 4, 7, 10, 14, 16, 19, 22,
    1, 3, 7, 11, 14, 15, 19, 21,
    2, 3, 7,  9, 13, 16, 20, 22,
    2, 3, 6,  9, 12, 16, 20, 21,
    1, 5, 6, 10, 13, 16, 20, 21,
    2, 5, 8,  9, 13, 17, 19, 22,
    2, 4, 6, 10, 14, 17, 19, 21,
    2, 5, 7,  9, 13, 16, 19, 22,
    0, 3, 8, 10, 12, 16, 19, 22,
    1, 3, 7, 10, 13, 16, 19, 21,
    1, 3, 8, 10, 13, 17, 20, 22,
    2, 4, 6,  9, 14, 15, 19, 22,
    1, 4, 6,  9, 13, 16, 19, 21,
    2, 4, 8, 10, 14, 15, 19, 22,
    1, 4, 7, 10, 14, 16, 19, 21,
  };
  for (size_t i = 0; i < kNRows * kNCols; ++i) {
    ASSERT_EQ(solution[i], gidx[i]);
  }
}

TEST(GpuHist, BuildGidxSparse) {
  int constexpr kNRows = 16, kNCols = 8;
  TrainParam param;
  param.max_depth = 1;
  param.n_gpus = 1;
  param.max_leaves = 0;

  DeviceShard<GradientPairPrecise> shard(0, 0, 0, kNRows, param, kNCols);
  BuildGidx(&shard, kNRows, kNCols, 0.9f);

  std::vector<common::CompressedByteT> h_gidx_buffer(shard.gidx_buffer.size());
  dh::CopyDeviceSpanToVector(&h_gidx_buffer, shard.gidx_buffer);
  common::CompressedIterator<uint32_t> gidx(h_gidx_buffer.data(), 25);

  ASSERT_LE(shard.ellpack_matrix.row_stride, 3);

  // row_stride = 3, 16 rows, 48 entries for ELLPack
  std::vector<uint32_t> solution = {
    15, 24, 24,  0, 24, 24, 24, 24, 24, 24, 24, 24, 20, 24, 24, 24,
    24, 24, 24, 24, 24,  5, 24, 24,  0, 16, 24, 15, 24, 24, 24, 24,
    24,  7, 14, 16,  4, 24, 24, 24, 24, 24,  9, 24, 24,  1, 24, 24
  };
  for (size_t i = 0; i < kNRows * shard.ellpack_matrix.row_stride; ++i) {
    ASSERT_EQ(solution[i], gidx[i]);
  }
}

std::vector<GradientPairPrecise> GetHostHistGpair() {
  // 24 bins, 3 bins for each feature (column).
  std::vector<GradientPairPrecise> hist_gpair = {
    {0.8314f, 0.7147f}, {1.7989f, 3.7312f}, {3.3846f, 3.4598f},
    {2.9277f, 3.5886f}, {1.8429f, 2.4152f}, {1.2443f, 1.9019f},
    {1.6380f, 2.9174f}, {1.5657f, 2.5107f}, {2.8111f, 2.4776f},
    {2.1322f, 3.0651f}, {3.2927f, 3.8540f}, {0.5899f, 0.9866f},
    {1.5185f, 1.6263f}, {2.0686f, 3.1844f}, {2.4278f, 3.0950f},
    {1.5105f, 2.1403f}, {2.6922f, 4.2217f}, {1.8122f, 1.5437f},
    {0.0000f, 0.0000f}, {4.3245f, 5.7955f}, {1.6903f, 2.1103f},
    {2.4012f, 4.4754f}, {3.6136f, 3.4303f}, {0.0000f, 0.0000f}
  };
  return hist_gpair;
}

template <typename GradientSumT>
void TestBuildHist(GPUHistBuilderBase<GradientSumT>& builder) {
  int const kNRows = 16, kNCols = 8;

  TrainParam param;
  param.max_depth = 6;
  param.n_gpus = 1;
  param.max_leaves = 0;

  DeviceShard<GradientSumT> shard(0, 0, 0, kNRows, param, kNCols);

  BuildGidx(&shard, kNRows, kNCols);

  xgboost::SimpleLCG gen;
  xgboost::SimpleRealUniformDistribution<bst_float> dist(0.0f, 1.0f);
  std::vector<GradientPair> h_gpair(kNRows);
  for (auto &gpair : h_gpair) {
    bst_float grad = dist(&gen);
    bst_float hess = dist(&gen);
    gpair = GradientPair(grad, hess);
  }

  int num_symbols = shard.n_bins + 1;

  thrust::host_vector<common::CompressedByteT> h_gidx_buffer (
      shard.gidx_buffer.size());

  common::CompressedByteT* d_gidx_buffer_ptr = shard.gidx_buffer.data();
  dh::safe_cuda(hipMemcpy(h_gidx_buffer.data(), d_gidx_buffer_ptr,
                           sizeof(common::CompressedByteT) * shard.gidx_buffer.size(),
                           hipMemcpyDeviceToHost));
  auto gidx = common::CompressedIterator<uint32_t>(h_gidx_buffer.data(),
                                                   num_symbols);

  shard.ridx_segments.resize(1);
  shard.ridx_segments[0] = Segment(0, kNRows);
  shard.hist.AllocateHistogram(0);
  dh::CopyVectorToDeviceSpan(shard.gpair, h_gpair);
  thrust::sequence(
      thrust::device_pointer_cast(shard.ridx.Current()),
      thrust::device_pointer_cast(shard.ridx.Current() + shard.ridx.Size()));

  builder.Build(&shard, 0);
  DeviceHistogram<GradientSumT> d_hist = shard.hist;

  auto node_histogram = d_hist.GetNodeHistogram(0);
  // d_hist.data stored in float, not gradient pair
  thrust::host_vector<GradientSumT> h_result (d_hist.Data().size() / 2);
  size_t data_size =
      sizeof(GradientSumT) /
      (sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT));
  data_size *= d_hist.Data().size();
  dh::safe_cuda(hipMemcpy(h_result.data(), node_histogram.data(), data_size,
                           hipMemcpyDeviceToHost));

  std::vector<GradientPairPrecise> solution = GetHostHistGpair();
  std::cout << std::fixed;
  for (size_t i = 0; i < h_result.size(); ++i) {
    EXPECT_NEAR(h_result[i].GetGrad(), solution[i].GetGrad(), 0.01f);
    EXPECT_NEAR(h_result[i].GetHess(), solution[i].GetHess(), 0.01f);
  }
}

TEST(GpuHist, BuildHistGlobalMem) {
  GlobalMemHistBuilder<GradientPairPrecise> double_builder;
  TestBuildHist(double_builder);
  GlobalMemHistBuilder<GradientPair> float_builder;
  TestBuildHist(float_builder);
}

TEST(GpuHist, BuildHistSharedMem) {
  SharedMemHistBuilder<GradientPairPrecise> double_builder;
  TestBuildHist(double_builder);
  SharedMemHistBuilder<GradientPair> float_builder;
  TestBuildHist(float_builder);
}

common::HistCutMatrix GetHostCutMatrix () {
  common::HistCutMatrix cmat;
  cmat.row_ptr = {0, 3, 6, 9, 12, 15, 18, 21, 24};
  cmat.min_val = {0.1f, 0.2f, 0.3f, 0.1f, 0.2f, 0.3f, 0.2f, 0.2f};
  // 24 cut fields, 3 cut fields for each feature (column).
  // Each row of the cut represents the cuts for a data column.
  cmat.cut = {0.30f, 0.67f, 1.64f,
              0.32f, 0.77f, 1.95f,
              0.29f, 0.70f, 1.80f,
              0.32f, 0.75f, 1.85f,
              0.18f, 0.59f, 1.69f,
              0.25f, 0.74f, 2.00f,
              0.26f, 0.74f, 1.98f,
              0.26f, 0.71f, 1.83f};
  return cmat;
}

// TODO(trivialfis): This test is over simplified.
TEST(GpuHist, EvaluateSplits) {
  constexpr int kNRows = 16;
  constexpr int kNCols = 8;

  TrainParam param;
  param.max_depth = 1;
  param.n_gpus = 1;
  param.colsample_bynode = 1;
  param.colsample_bylevel = 1;
  param.colsample_bytree = 1;
  param.min_child_weight = 0.01;

  // Disable all parameters.
  param.reg_alpha = 0.0;
  param.reg_lambda = 0;
  param.max_delta_step = 0.0;

  for (size_t i = 0; i < kNCols; ++i) {
    param.monotone_constraints.emplace_back(0);
  }

  int max_bins = 4;

  // Initialize DeviceShard
  std::unique_ptr<DeviceShard<GradientPairPrecise>> shard{
      new DeviceShard<GradientPairPrecise>(0, 0, 0, kNRows, param, kNCols)};
  // Initialize DeviceShard::node_sum_gradients
  shard->node_sum_gradients = {{6.4f, 12.8f}};

  // Initialize DeviceShard::cut
  common::HistCutMatrix cmat = GetHostCutMatrix();

  // Copy cut matrix to device.
  shard->ba.Allocate(0,
                     &(shard->feature_segments), cmat.row_ptr.size(),
                     &(shard->min_fvalue), cmat.min_val.size(),
                     &(shard->gidx_fvalue_map), 24,
                     &(shard->monotone_constraints), kNCols);
  dh::CopyVectorToDeviceSpan(shard->feature_segments, cmat.row_ptr);
  dh::CopyVectorToDeviceSpan(shard->gidx_fvalue_map, cmat.cut);
  dh::CopyVectorToDeviceSpan(shard->monotone_constraints,
                             param.monotone_constraints);
  shard->ellpack_matrix.feature_segments = shard->feature_segments;
  shard->ellpack_matrix.gidx_fvalue_map = shard->gidx_fvalue_map;
  dh::CopyVectorToDeviceSpan(shard->min_fvalue, cmat.min_val);
  shard->ellpack_matrix.min_fvalue = shard->min_fvalue;

  // Initialize DeviceShard::hist
  shard->hist.Init(0, (max_bins - 1) * kNCols);
  shard->hist.AllocateHistogram(0);
  // Each row of hist_gpair represents gpairs for one feature.
  // Each entry represents a bin.
  std::vector<GradientPairPrecise> hist_gpair = GetHostHistGpair();
  std::vector<bst_float> hist;
  for (auto pair : hist_gpair) {
    hist.push_back(pair.GetGrad());
    hist.push_back(pair.GetHess());
  }

  ASSERT_EQ(shard->hist.Data().size(), hist.size());
  thrust::copy(hist.begin(), hist.end(),
               shard->hist.Data().begin());

  shard->column_sampler.Init(kNCols,
                                  param.colsample_bynode,
                                  param.colsample_bylevel,
                                  param.colsample_bytree,
                                  false);

  RegTree tree;
  MetaInfo info;
  info.num_row_ = kNRows;
  info.num_col_ = kNCols;

  shard->node_value_constraints.resize(1);
  shard->node_value_constraints[0].lower_bound = -1.0;
  shard->node_value_constraints[0].upper_bound = 1.0;

  std::vector<DeviceSplitCandidate> res =
    shard->EvaluateSplits({ 0,0 }, tree, kNCols);

  ASSERT_EQ(res[0].findex, 7);
  ASSERT_EQ(res[1].findex, 7);
  ASSERT_NEAR(res[0].fvalue, 0.26, xgboost::kRtEps);
  ASSERT_NEAR(res[1].fvalue, 0.26, xgboost::kRtEps);
}

TEST(GpuHist, ApplySplit) {
  GPUHistMakerSpecialised<GradientPairPrecise> hist_maker =
      GPUHistMakerSpecialised<GradientPairPrecise>();
  int constexpr kNId = 0;
  int constexpr kNRows = 16;
  int constexpr kNCols = 8;

  TrainParam param;
  std::vector<std::pair<std::string, std::string>> args = {};
  param.InitAllowUnknown(args);

  // Initialize shard
  for (size_t i = 0; i < kNCols; ++i) {
    param.monotone_constraints.emplace_back(0);
  }

  hist_maker.shards_.resize(1);
  hist_maker.shards_[0].reset(
      new DeviceShard<GradientPairPrecise>(0, 0, 0, kNRows, param, kNCols));

  auto& shard = hist_maker.shards_.at(0);
  shard->ridx_segments.resize(3);  // 3 nodes.
  shard->node_sum_gradients.resize(3);

  shard->ridx_segments[0] = Segment(0, kNRows);
  shard->ba.Allocate(0, &(shard->ridx), kNRows,
                     &(shard->position), kNRows);
  shard->ellpack_matrix.row_stride = kNCols;
  thrust::sequence(
      thrust::device_pointer_cast(shard->ridx.Current()),
      thrust::device_pointer_cast(shard->ridx.Current() + shard->ridx.Size()));
  // Initialize GPUHistMaker
  hist_maker.param_ = param;
  RegTree tree;

  DeviceSplitCandidate candidate;
  candidate.Update(2, kLeftDir,
                   0.59, 4,  // fvalue has to be equal to one of the cut field
                   GradientPair(8.2, 2.8), GradientPair(6.3, 3.6),
                   GPUTrainingParam(param));
  ExpandEntry candidate_entry {0, 0, candidate, 0};
  candidate_entry.nid = kNId;

  // Used to get bin_id in update position.
  common::HistCutMatrix cmat = GetHostCutMatrix();
  hist_maker.hmat_ = cmat;

  MetaInfo info;
  info.num_row_ = kNRows;
  info.num_col_ = kNCols;
  info.num_nonzero_ = kNRows * kNCols;  // Dense

  // Initialize gidx
  int n_bins = 24;
  int row_stride = kNCols;
  int num_symbols = n_bins + 1;
  size_t compressed_size_bytes =
      common::CompressedBufferWriter::CalculateBufferSize(row_stride * kNRows,
                                                          num_symbols);
  shard->ba.Allocate(0, &(shard->gidx_buffer), compressed_size_bytes,
                     &(shard->feature_segments), cmat.row_ptr.size(),
                     &(shard->min_fvalue), cmat.min_val.size(),
                     &(shard->gidx_fvalue_map), 24);
  dh::CopyVectorToDeviceSpan(shard->feature_segments, cmat.row_ptr);
  dh::CopyVectorToDeviceSpan(shard->gidx_fvalue_map, cmat.cut);
  shard->ellpack_matrix.feature_segments = shard->feature_segments;
  shard->ellpack_matrix.gidx_fvalue_map = shard->gidx_fvalue_map;
  dh::CopyVectorToDeviceSpan(shard->min_fvalue, cmat.min_val);
  shard->ellpack_matrix.min_fvalue = shard->min_fvalue;
  shard->ellpack_matrix.is_dense = true;

  common::CompressedBufferWriter wr(num_symbols);
  // gidx 14 should go right, 12 goes left
  std::vector<int> h_gidx (kNRows * row_stride, 14);
  h_gidx[4] = 12;
  h_gidx[12] = 12;
  std::vector<common::CompressedByteT> h_gidx_compressed (compressed_size_bytes);

  wr.Write(h_gidx_compressed.data(), h_gidx.begin(), h_gidx.end());
  dh::CopyVectorToDeviceSpan(shard->gidx_buffer, h_gidx_compressed);

  shard->ellpack_matrix.gidx_iter = common::CompressedIterator<uint32_t>(
      shard->gidx_buffer.data(), num_symbols);

  hist_maker.info_ = &info;
  shard->ApplySplit(candidate_entry, &tree);
  shard->UpdatePosition(candidate_entry.nid, tree[candidate_entry.nid]);

  ASSERT_FALSE(tree[kNId].IsLeaf());

  int left_nidx = tree[kNId].LeftChild();
  int right_nidx = tree[kNId].RightChild();

  ASSERT_EQ(shard->ridx_segments[left_nidx].begin, 0);
  ASSERT_EQ(shard->ridx_segments[left_nidx].end, 2);
  ASSERT_EQ(shard->ridx_segments[right_nidx].begin, 2);
  ASSERT_EQ(shard->ridx_segments[right_nidx].end, 16);
}

void TestSortPosition(const std::vector<int>& position_in, int left_idx,
                      int right_idx) {
  std::vector<int64_t> left_count = {
      std::count(position_in.begin(), position_in.end(), left_idx)};
  thrust::device_vector<int64_t> d_left_count = left_count;
  thrust::device_vector<int> position = position_in;
  thrust::device_vector<int> position_out(position.size());

  thrust::device_vector<bst_uint> ridx(position.size());
  thrust::sequence(ridx.begin(), ridx.end());
  thrust::device_vector<bst_uint> ridx_out(ridx.size());
  dh::CubMemory tmp;
  SortPosition(
      &tmp, common::Span<int>(position.data().get(), position.size()),
      common::Span<int>(position_out.data().get(), position_out.size()),
      common::Span<bst_uint>(ridx.data().get(), ridx.size()),
      common::Span<bst_uint>(ridx_out.data().get(), ridx_out.size()), left_idx,
      right_idx, d_left_count.data().get(), nullptr);
  thrust::host_vector<int> position_result = position_out;
  thrust::host_vector<int> ridx_result = ridx_out;

  // Check position is sorted
  EXPECT_TRUE(std::is_sorted(position_result.begin(), position_result.end()));
  // Check row indices are sorted inside left and right segment
  EXPECT_TRUE(
      std::is_sorted(ridx_result.begin(), ridx_result.begin() + left_count[0]));
  EXPECT_TRUE(
      std::is_sorted(ridx_result.begin() + left_count[0], ridx_result.end()));

  // Check key value pairs are the same
  for (auto i = 0ull; i < ridx_result.size(); i++) {
    EXPECT_EQ(position_result[i], position_in[ridx_result[i]]);
  }
}

TEST(GpuHist, SortPosition) {
  TestSortPosition({1, 2, 1, 2, 1}, 1, 2);
  TestSortPosition({1, 1, 1, 1}, 1, 2);
  TestSortPosition({2, 2, 2, 2}, 1, 2);
  TestSortPosition({1, 2, 1, 2, 3}, 1, 2);
}

TEST(GpuHist, TestHistogramIndex) {
  // Test if the compressed histogram index matches when using a sparse
  // dmatrix with and without using external memory

  int constexpr kNRows = 1000, kNCols = 10;

  // Build 2 matrices and build a histogram maker with that
  tree::GPUHistMakerSpecialised<GradientPairPrecise> hist_maker, hist_maker_ext;
  std::unique_ptr<DMatrix> hist_maker_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 0, true));
  std::unique_ptr<DMatrix> hist_maker_ext_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 128UL, true));

  std::vector<std::pair<std::string, std::string>> training_params = {
    {"max_depth", "1"},
    {"max_leaves", "0"},
    {"n_gpus", "1"}
  };

  hist_maker.Init(training_params);
  hist_maker.InitDataOnce(hist_maker_dmat.get());
  hist_maker_ext.Init(training_params);
  hist_maker_ext.InitDataOnce(hist_maker_ext_dmat.get());

  // Extract the device shards from the histogram makers and from that its compressed
  // histogram index
  const auto &dev_shard = hist_maker.shards_[0];
  std::vector<common::CompressedByteT> h_gidx_buffer(dev_shard->gidx_buffer.size());
  dh::CopyDeviceSpanToVector(&h_gidx_buffer, dev_shard->gidx_buffer);

  const auto &dev_shard_ext = hist_maker_ext.shards_[0];
  std::vector<common::CompressedByteT> h_gidx_buffer_ext(dev_shard_ext->gidx_buffer.size());
  dh::CopyDeviceSpanToVector(&h_gidx_buffer_ext, dev_shard_ext->gidx_buffer);

  ASSERT_EQ(dev_shard->n_bins, dev_shard_ext->n_bins);
  ASSERT_EQ(dev_shard->gidx_buffer.size(), dev_shard_ext->gidx_buffer.size());

  ASSERT_EQ(h_gidx_buffer, h_gidx_buffer_ext);
}

}  // namespace tree
}  // namespace xgboost
