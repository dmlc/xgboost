/*!
 * Copyright 2017-2019 XGBoost contributors
 */
#include <thrust/device_vector.h>
#include <dmlc/filesystem.h>
#include <xgboost/base.h>
#include <random>
#include <string>
#include <vector>

#include "../helpers.h"
#include "gtest/gtest.h"

#include "xgboost/json.h"
#include "../../../src/data/sparse_page_source.h"
#include "../../../src/tree/updater_gpu_hist.cu"
#include "../../../src/tree/updater_gpu_common.cuh"
#include "../../../src/common/common.h"
#include "../../../src/tree/constraints.cuh"

namespace xgboost {
namespace tree {

TEST(GpuHist, DeviceHistogram) {
  // Ensures that node allocates correctly after reaching `kStopGrowingSize`.
  dh::SaveCudaContext{
    [&]() {
      dh::safe_cuda(hipSetDevice(0));
      constexpr size_t kNBins = 128;
      constexpr size_t kNNodes = 4;
      constexpr size_t kStopGrowing = kNNodes * kNBins * 2u;
      DeviceHistogram<GradientPairPrecise, kStopGrowing> histogram;
      histogram.Init(0, kNBins);
      for (size_t i = 0; i < kNNodes; ++i) {
        histogram.AllocateHistogram(i);
      }
      histogram.Reset();
      ASSERT_EQ(histogram.Data().size(), kStopGrowing);

      // Use allocated memory but do not erase nidx_map.
      for (size_t i = 0; i < kNNodes; ++i) {
        histogram.AllocateHistogram(i);
      }
      for (size_t i = 0; i < kNNodes; ++i) {
        ASSERT_TRUE(histogram.HistogramExists(i));
      }

      // Erase existing nidx_map.
      for (size_t i = kNNodes; i < kNNodes * 2; ++i) {
        histogram.AllocateHistogram(i);
      }
      for (size_t i = 0; i < kNNodes; ++i) {
        ASSERT_FALSE(histogram.HistogramExists(i));
      }
    }
  };
}

std::vector<GradientPairPrecise> GetHostHistGpair() {
  // 24 bins, 3 bins for each feature (column).
  std::vector<GradientPairPrecise> hist_gpair = {
    {0.8314f, 0.7147f}, {1.7989f, 3.7312f}, {3.3846f, 3.4598f},
    {2.9277f, 3.5886f}, {1.8429f, 2.4152f}, {1.2443f, 1.9019f},
    {1.6380f, 2.9174f}, {1.5657f, 2.5107f}, {2.8111f, 2.4776f},
    {2.1322f, 3.0651f}, {3.2927f, 3.8540f}, {0.5899f, 0.9866f},
    {1.5185f, 1.6263f}, {2.0686f, 3.1844f}, {2.4278f, 3.0950f},
    {1.5105f, 2.1403f}, {2.6922f, 4.2217f}, {1.8122f, 1.5437f},
    {0.0000f, 0.0000f}, {4.3245f, 5.7955f}, {1.6903f, 2.1103f},
    {2.4012f, 4.4754f}, {3.6136f, 3.4303f}, {0.0000f, 0.0000f}
  };
  return hist_gpair;
}

template <typename GradientSumT>
void TestBuildHist(bool use_shared_memory_histograms) {
  int const kNRows = 16, kNCols = 8;

  TrainParam param;
  std::vector<std::pair<std::string, std::string>> args {
    {"max_depth", "6"},
    {"max_leaves", "0"},
  };
  param.Init(args);
  auto page = BuildEllpackPage(kNRows, kNCols);
  BatchParam batch_param{};
  GPUHistMakerDevice<GradientSumT> maker(0, page.get(), kNRows, param, kNCols, kNCols, batch_param);
  maker.InitHistogram();

  xgboost::SimpleLCG gen;
  xgboost::SimpleRealUniformDistribution<bst_float> dist(0.0f, 1.0f);
  std::vector<GradientPair> h_gpair(kNRows);
  for (auto &gpair : h_gpair) {
    bst_float grad = dist(&gen);
    bst_float hess = dist(&gen);
    gpair = GradientPair(grad, hess);
  }

  thrust::host_vector<common::CompressedByteT> h_gidx_buffer (page->gidx_buffer.size());

  common::CompressedByteT* d_gidx_buffer_ptr = page->gidx_buffer.data();
  dh::safe_cuda(hipMemcpy(h_gidx_buffer.data(), d_gidx_buffer_ptr,
                           sizeof(common::CompressedByteT) * page->gidx_buffer.size(),
                           hipMemcpyDeviceToHost));

  maker.row_partitioner.reset(new RowPartitioner(0, kNRows));
  maker.hist.AllocateHistogram(0);
  dh::CopyVectorToDeviceSpan(maker.gpair, h_gpair);

  maker.use_shared_memory_histograms = use_shared_memory_histograms;
  maker.BuildHist(0);
  DeviceHistogram<GradientSumT> d_hist = maker.hist;

  auto node_histogram = d_hist.GetNodeHistogram(0);
  // d_hist.data stored in float, not gradient pair
  thrust::host_vector<GradientSumT> h_result (d_hist.Data().size() / 2);
  size_t data_size =
      sizeof(GradientSumT) /
      (sizeof(GradientSumT) / sizeof(typename GradientSumT::ValueT));
  data_size *= d_hist.Data().size();
  dh::safe_cuda(hipMemcpy(h_result.data(), node_histogram.data(), data_size,
                           hipMemcpyDeviceToHost));

  std::vector<GradientPairPrecise> solution = GetHostHistGpair();
  std::cout << std::fixed;
  for (size_t i = 0; i < h_result.size(); ++i) {
    EXPECT_NEAR(h_result[i].GetGrad(), solution[i].GetGrad(), 0.01f);
    EXPECT_NEAR(h_result[i].GetHess(), solution[i].GetHess(), 0.01f);
  }
}

TEST(GpuHist, BuildHistGlobalMem) {
  TestBuildHist<GradientPairPrecise>(false);
  TestBuildHist<GradientPair>(false);
}

TEST(GpuHist, BuildHistSharedMem) {
  TestBuildHist<GradientPairPrecise>(true);
  TestBuildHist<GradientPair>(true);
}

HistogramCutsWrapper GetHostCutMatrix () {
  HistogramCutsWrapper cmat;
  cmat.SetPtrs({0, 3, 6, 9, 12, 15, 18, 21, 24});
  cmat.SetMins({0.1f, 0.2f, 0.3f, 0.1f, 0.2f, 0.3f, 0.2f, 0.2f});
  // 24 cut fields, 3 cut fields for each feature (column).
  // Each row of the cut represents the cuts for a data column.
  cmat.SetValues({0.30f, 0.67f, 1.64f,
              0.32f, 0.77f, 1.95f,
              0.29f, 0.70f, 1.80f,
              0.32f, 0.75f, 1.85f,
              0.18f, 0.59f, 1.69f,
              0.25f, 0.74f, 2.00f,
              0.26f, 0.74f, 1.98f,
              0.26f, 0.71f, 1.83f});
  return cmat;
}

// TODO(trivialfis): This test is over simplified.
TEST(GpuHist, EvaluateSplits) {
  constexpr int kNRows = 16;
  constexpr int kNCols = 8;

  TrainParam param;

  std::vector<std::pair<std::string, std::string>> args {
    {"max_depth", "1"},
    {"max_leaves", "0"},

    // Disable all other parameters.
    {"colsample_bynode", "1"},
    {"colsample_bylevel", "1"},
    {"colsample_bytree", "1"},
    {"min_child_weight", "0.01"},
    {"reg_alpha", "0"},
    {"reg_lambda", "0"},
    {"max_delta_step", "0"}
  };
  param.Init(args);
  for (size_t i = 0; i < kNCols; ++i) {
    param.monotone_constraints.emplace_back(0);
  }

  int max_bins = 4;

  // Initialize GPUHistMakerDevice
  auto page = BuildEllpackPage(kNRows, kNCols);
  BatchParam batch_param{};
  GPUHistMakerDevice<GradientPairPrecise>
      maker(0, page.get(), kNRows, param, kNCols, kNCols, batch_param);
  // Initialize GPUHistMakerDevice::node_sum_gradients
  maker.node_sum_gradients = {{6.4f, 12.8f}};

  // Initialize GPUHistMakerDevice::cut
  auto cmat = GetHostCutMatrix();

  // Copy cut matrix to device.
  maker.ba.Allocate(0,
                    &(page->matrix.info.feature_segments), cmat.Ptrs().size(),
                    &(page->matrix.info.min_fvalue), cmat.MinValues().size(),
                    &(page->matrix.info.gidx_fvalue_map), 24,
                    &(maker.monotone_constraints), kNCols);
  dh::CopyVectorToDeviceSpan(page->matrix.info.feature_segments, cmat.Ptrs());
  dh::CopyVectorToDeviceSpan(page->matrix.info.gidx_fvalue_map, cmat.Values());
  dh::CopyVectorToDeviceSpan(maker.monotone_constraints, param.monotone_constraints);
  dh::CopyVectorToDeviceSpan(page->matrix.info.min_fvalue, cmat.MinValues());

  // Initialize GPUHistMakerDevice::hist
  maker.hist.Init(0, (max_bins - 1) * kNCols);
  maker.hist.AllocateHistogram(0);
  // Each row of hist_gpair represents gpairs for one feature.
  // Each entry represents a bin.
  std::vector<GradientPairPrecise> hist_gpair = GetHostHistGpair();
  std::vector<bst_float> hist;
  for (auto pair : hist_gpair) {
    hist.push_back(pair.GetGrad());
    hist.push_back(pair.GetHess());
  }

  ASSERT_EQ(maker.hist.Data().size(), hist.size());
  thrust::copy(hist.begin(), hist.end(),
               maker.hist.Data().begin());

  maker.column_sampler.Init(kNCols,
                            param.colsample_bynode,
                            param.colsample_bylevel,
                            param.colsample_bytree,
                            false);

  RegTree tree;
  MetaInfo info;
  info.num_row_ = kNRows;
  info.num_col_ = kNCols;

  maker.node_value_constraints.resize(1);
  maker.node_value_constraints[0].lower_bound = -1.0;
  maker.node_value_constraints[0].upper_bound = 1.0;

  std::vector<DeviceSplitCandidate> res = maker.EvaluateSplits({0, 0 }, tree, kNCols);

  ASSERT_EQ(res[0].findex, 7);
  ASSERT_EQ(res[1].findex, 7);
  ASSERT_NEAR(res[0].fvalue, 0.26, xgboost::kRtEps);
  ASSERT_NEAR(res[1].fvalue, 0.26, xgboost::kRtEps);
}

void TestHistogramIndexImpl() {
  // Test if the compressed histogram index matches when using a sparse
  // dmatrix with and without using external memory

  int constexpr kNRows = 1000, kNCols = 10;

  // Build 2 matrices and build a histogram maker with that
  tree::GPUHistMakerSpecialised<GradientPairPrecise> hist_maker, hist_maker_ext;
  std::unique_ptr<DMatrix> hist_maker_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 0, true));

  dmlc::TemporaryDirectory tempdir;
  std::unique_ptr<DMatrix> hist_maker_ext_dmat(
    CreateSparsePageDMatrixWithRC(kNRows, kNCols, 128UL, true, tempdir));

  std::vector<std::pair<std::string, std::string>> training_params = {
    {"max_depth", "10"},
    {"max_leaves", "0"}
  };

  GenericParameter generic_param(CreateEmptyGenericParam(0));
  hist_maker.Configure(training_params, &generic_param);
  hist_maker.InitDataOnce(hist_maker_dmat.get());
  hist_maker_ext.Configure(training_params, &generic_param);
  hist_maker_ext.InitDataOnce(hist_maker_ext_dmat.get());

  // Extract the device maker from the histogram makers and from that its compressed
  // histogram index
  const auto &maker = hist_maker.maker;
  std::vector<common::CompressedByteT> h_gidx_buffer(maker->page->gidx_buffer.size());
  dh::CopyDeviceSpanToVector(&h_gidx_buffer, maker->page->gidx_buffer);

  const auto &maker_ext = hist_maker_ext.maker;
  std::vector<common::CompressedByteT> h_gidx_buffer_ext(maker_ext->page->gidx_buffer.size());
  dh::CopyDeviceSpanToVector(&h_gidx_buffer_ext, maker_ext->page->gidx_buffer);

  ASSERT_EQ(maker->page->matrix.info.n_bins, maker_ext->page->matrix.info.n_bins);
  ASSERT_EQ(maker->page->gidx_buffer.size(), maker_ext->page->gidx_buffer.size());

  ASSERT_EQ(h_gidx_buffer, h_gidx_buffer_ext);
}

TEST(GpuHist, TestHistogramIndex) {
  TestHistogramIndexImpl();
}

// gamma is an alias of min_split_loss
int32_t TestMinSplitLoss(DMatrix* dmat, float gamma, HostDeviceVector<GradientPair>* gpair) {
  Args args {
    {"max_depth", "1"},
    {"max_leaves", "0"},

    // Disable all other parameters.
    {"colsample_bynode", "1"},
    {"colsample_bylevel", "1"},
    {"colsample_bytree", "1"},
    {"min_child_weight", "0.01"},
    {"reg_alpha", "0"},
    {"reg_lambda", "0"},
    {"max_delta_step", "0"},

    // test gamma
    {"gamma", std::to_string(gamma)}
  };

  tree::GPUHistMakerSpecialised<GradientPairPrecise> hist_maker;
  GenericParameter generic_param(CreateEmptyGenericParam(0));
  hist_maker.Configure(args, &generic_param);

  RegTree tree;
  hist_maker.Update(gpair, dmat, {&tree});

  auto n_nodes = tree.NumExtraNodes();
  return n_nodes;
}

HostDeviceVector<GradientPair> GenerateRandomGradients(const size_t n_rows) {
  xgboost::SimpleLCG gen;
  xgboost::SimpleRealUniformDistribution<bst_float> dist(0.0f, 1.0f);
  std::vector<GradientPair> h_gpair(n_rows);
  for (auto &gpair : h_gpair) {
    bst_float grad = dist(&gen);
    bst_float hess = dist(&gen);
    gpair = GradientPair(grad, hess);
  }
  HostDeviceVector<GradientPair> gpair(h_gpair);
  return gpair;
}

TEST(GpuHist, MinSplitLoss) {
  constexpr size_t kRows = 32;
  constexpr size_t kCols = 16;
  constexpr float kSparsity = 0.6;
  auto dmat = CreateDMatrix(kRows, kCols, kSparsity, 3);
  auto gpair = GenerateRandomGradients(kRows);

  {
    int32_t n_nodes = TestMinSplitLoss((*dmat).get(), 0.01, &gpair);
    // This is not strictly verified, meaning the numeber `2` is whatever GPU_Hist retured
    // when writing this test, and only used for testing larger gamma (below) does prevent
    // building tree.
    ASSERT_EQ(n_nodes, 2);
  }
  {
    int32_t n_nodes = TestMinSplitLoss((*dmat).get(), 100.0, &gpair);
    // No new nodes with gamma == 100.
    ASSERT_EQ(n_nodes, static_cast<decltype(n_nodes)>(0));
  }
  delete dmat;
}

void UpdateTree(HostDeviceVector<GradientPair>* gpair,
                DMatrix* dmat,
                size_t gpu_page_size,
                RegTree* tree,
                HostDeviceVector<bst_float>* preds) {
  constexpr size_t kMaxBin = 2;

  if (gpu_page_size > 0) {
    // Loop over the batches and count the records
    int64_t batch_count = 0;
    int64_t row_count = 0;
    for (const auto& batch : dmat->GetBatches<EllpackPage>({0, kMaxBin, 0, gpu_page_size})) {
      EXPECT_LT(batch.Size(), dmat->Info().num_row_);
      batch_count++;
      row_count += batch.Size();
    }
    EXPECT_GE(batch_count, 2);
    EXPECT_EQ(row_count, dmat->Info().num_row_);
  }

  Args args{
      {"max_depth", "2"},
      {"max_bin", std::to_string(kMaxBin)},
      {"min_child_weight", "0.0"},
      {"reg_alpha", "0"},
      {"reg_lambda", "0"}
  };

  tree::GPUHistMakerSpecialised<GradientPairPrecise> hist_maker;
  GenericParameter generic_param(CreateEmptyGenericParam(0));
  generic_param.gpu_page_size = gpu_page_size;
  hist_maker.Configure(args, &generic_param);

  hist_maker.Update(gpair, dmat, {tree});
  hist_maker.UpdatePredictionCache(dmat, preds);
}

TEST(GpuHist, ExternalMemory) {
  constexpr size_t kRows = 6;
  constexpr size_t kCols = 2;
  constexpr size_t kPageSize = 1;

  // Create an in-memory DMatrix.
  std::unique_ptr<DMatrix> dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, 0, true));

  // Create a DMatrix with multiple batches.
  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix>
      dmat_ext(CreateSparsePageDMatrixWithRC(kRows, kCols, kPageSize, true, tmpdir));

  auto gpair = GenerateRandomGradients(kRows);

  // Build a tree using the in-memory DMatrix.
  RegTree tree;
  HostDeviceVector<bst_float> preds(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat.get(), 0, &tree, &preds);

  // Build another tree using multiple ELLPACK pages.
  RegTree tree_ext;
  HostDeviceVector<bst_float> preds_ext(kRows, 0.0, 0);
  UpdateTree(&gpair, dmat_ext.get(), kPageSize, &tree_ext, &preds_ext);

  // Make sure the predictions are the same.
  auto preds_h = preds.ConstHostVector();
  auto preds_ext_h = preds_ext.ConstHostVector();
  for (int i = 0; i < kRows; i++) {
    ASSERT_FLOAT_EQ(preds_h[i], preds_ext_h[i]);
  }
}

TEST(GpuHist, Config_IO) {
  GenericParameter generic_param(CreateEmptyGenericParam(0));
  std::unique_ptr<TreeUpdater> updater {TreeUpdater::Create("grow_gpu_hist", &generic_param) };
  updater->Configure(Args{});

  Json j_updater { Object() };
  updater->SaveConfig(&j_updater);
  ASSERT_TRUE(IsA<Object>(j_updater["gpu_hist_train_param"]));
  ASSERT_TRUE(IsA<Object>(j_updater["train_param"]));
  updater->LoadConfig(j_updater);

  Json j_updater_roundtrip { Object() };
  updater->SaveConfig(&j_updater_roundtrip);
  ASSERT_TRUE(IsA<Object>(j_updater_roundtrip["gpu_hist_train_param"]));
  ASSERT_TRUE(IsA<Object>(j_updater_roundtrip["train_param"]));

  ASSERT_EQ(j_updater, j_updater_roundtrip);
}

}  // namespace tree
}  // namespace xgboost
