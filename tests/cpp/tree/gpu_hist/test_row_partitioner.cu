#include <gtest/gtest.h>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include "../../../../src/tree/gpu_hist/row_partitioner.cuh"
#include "../../helpers.h"

namespace xgboost {
namespace tree {

void TestSortPosition(const std::vector<int>& position_in, int left_idx,
                      int right_idx) {
  dh::safe_cuda(hipSetDevice(0));
  std::vector<int64_t> left_count = {
      std::count(position_in.begin(), position_in.end(), left_idx)};
  dh::caching_device_vector<int64_t> d_left_count = left_count;
  dh::caching_device_vector<int> position = position_in;
  dh::caching_device_vector<int> position_out(position.size());

  dh::caching_device_vector<RowPartitioner::RowIndexT> ridx(position.size());
  thrust::sequence(ridx.begin(), ridx.end());
  dh::caching_device_vector<RowPartitioner::RowIndexT> ridx_out(ridx.size());
  RowPartitioner rp(0,10);
  rp.SortPosition(
      common::Span<int>(position.data().get(), position.size()),
      common::Span<int>(position_out.data().get(), position_out.size()),
      common::Span<RowPartitioner::RowIndexT>(ridx.data().get(), ridx.size()),
      common::Span<RowPartitioner::RowIndexT>(ridx_out.data().get(), ridx_out.size()), left_idx,
      right_idx, d_left_count.data().get(), nullptr);
  thrust::host_vector<int> position_result = position_out;
  thrust::host_vector<int> ridx_result = ridx_out;

  // Check position is sorted
  EXPECT_TRUE(std::is_sorted(position_result.begin(), position_result.end()));
  // Check row indices are sorted inside left and right segment
  EXPECT_TRUE(
      std::is_sorted(ridx_result.begin(), ridx_result.begin() + left_count[0]));
  EXPECT_TRUE(
      std::is_sorted(ridx_result.begin() + left_count[0], ridx_result.end()));

  // Check key value pairs are the same
  for (auto i = 0ull; i < ridx_result.size(); i++) {
    EXPECT_EQ(position_result[i], position_in[ridx_result[i]]);
  }
}
TEST(GpuHist, SortPosition) {
  TestSortPosition({1, 2, 1, 2, 1}, 1, 2);
  TestSortPosition({1, 1, 1, 1}, 1, 2);
  TestSortPosition({2, 2, 2, 2}, 1, 2);
  TestSortPosition({1, 2, 1, 2, 3}, 1, 2);
}

void TestUpdatePosition() {
  const int kNumRows = 10;
  RowPartitioner rp(0, kNumRows);
  auto rows = rp.GetRowsHost(0);
  EXPECT_EQ(rows.size(), kNumRows);
  for (auto i = 0ull; i < kNumRows; i++) {
    EXPECT_EQ(rows[i], i);
  }
  // Send the first five training instances to the right node
  // and the second 5 to the left node
  rp.UpdatePosition(0, 1, 2,
    [=] __device__(RowPartitioner::RowIndexT ridx) {
    if (ridx > 4) {
      return 1;
    }
    else {
      return 2;
    }
  });
  rows = rp.GetRowsHost(1);
  for (auto r : rows) {
    EXPECT_GT(r, 4);
  }
  rows = rp.GetRowsHost(2);
  for (auto r : rows) {
    EXPECT_LT(r, 5);
  }

  // Split the left node again
  rp.UpdatePosition(1, 3, 4, [=]__device__(RowPartitioner::RowIndexT ridx)
  {
    if (ridx < 7) {
      return 3
        ;
    }
    return 4;
  });
  EXPECT_EQ(rp.GetRows(3).size(), 2);
  EXPECT_EQ(rp.GetRows(4).size(), 3);
  // Check position is as expected
  EXPECT_EQ(rp.GetPositionHost(), std::vector<bst_node_t>({3,3,4,4,4,2,2,2,2,2}));
}

TEST(RowPartitioner, Basic) { TestUpdatePosition(); }

void TestFinalise() {
  const int kNumRows = 10;
  RowPartitioner rp(0, kNumRows);
  rp.FinalisePosition([=]__device__(RowPartitioner::RowIndexT ridx, int position)
  {
    return 7;
  });
  auto position = rp.GetPositionHost();
  for(auto p:position)
  {
    EXPECT_EQ(p, 7);
  }
}
TEST(RowPartitioner, Finalise) { TestFinalise(); }

void TestIncorrectRow() {
  RowPartitioner rp(0, 1);
  rp.UpdatePosition(0, 1, 2, [=]__device__ (RowPartitioner::RowIndexT ridx)
  {
    return 4; // This is not the left branch or the right branch
  });
}

TEST(RowPartitionerDeathTest, IncorrectRow) {
  ASSERT_DEATH({ TestIncorrectRow(); },".*");
}
}  // namespace tree
}  // namespace xgboost
