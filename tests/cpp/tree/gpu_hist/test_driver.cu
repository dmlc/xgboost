#include <gtest/gtest.h>
#include "../../../../src/tree/driver.h"
#include "../../../../src/tree/gpu_hist/expand_entry.cuh"

namespace xgboost {
namespace tree {

TEST(GpuHist, DriverDepthWise) {
  TrainParam p;
  p.UpdateAllowUnknown(Args{{"grow_policy", "depthwise"}});

  Driver<GPUExpandEntry> driver(p, 2);
  EXPECT_TRUE(driver.Pop().empty());
  DeviceSplitCandidate split;
  split.loss_chg = 1.0f;
  split.left_sum = {0.0f, 1.0f};
  split.right_sum = {0.0f, 1.0f};
  GPUExpandEntry root(0, 0, split, 2.0f, 1.0f, 1.0f);
  driver.Push({root});
  EXPECT_EQ(driver.Pop().front().nid, 0);
  driver.Push({GPUExpandEntry{1, 1, split, 2.0f, 1.0f, 1.0f}});
  driver.Push({GPUExpandEntry{2, 1, split, 2.0f, 1.0f, 1.0f}});
  driver.Push({GPUExpandEntry{3, 1, split, 2.0f, 1.0f, 1.0f}});
  driver.Push({GPUExpandEntry{4, 2, split, 2.0f, 1.0f, 1.0f}});
  // Should return 2 entries from level 1
  // as we limited the driver to pop maximum 2 nodes
  auto res = driver.Pop();
  EXPECT_EQ(res.size(), 2);
  for (auto &e : res) {
    EXPECT_EQ(e.depth, 1);
  }

  // Should now return 1 entry from level 1
  res = driver.Pop();
  EXPECT_EQ(res.size(), 1);
  EXPECT_EQ(res.at(0).depth, 1);

  res = driver.Pop();
  EXPECT_EQ(res.at(0).depth, 2);
  EXPECT_TRUE(driver.Pop().empty());
}

TEST(GpuHist, DriverLossGuided) {
  DeviceSplitCandidate high_gain;
  high_gain.left_sum = {0.0f, 1.0f};
  high_gain.right_sum = {0.0f, 1.0f};
  high_gain.loss_chg = 5.0f;
  DeviceSplitCandidate low_gain = high_gain;
  low_gain.loss_chg = 1.0f;

  TrainParam p;
  p.UpdateAllowUnknown(Args{{"grow_policy", "lossguide"}});

  Driver<GPUExpandEntry> driver(p);
  EXPECT_TRUE(driver.Pop().empty());
  GPUExpandEntry root(0, 0, high_gain, 2.0f, 1.0f, 1.0f );
  driver.Push({root});
  EXPECT_EQ(driver.Pop().front().nid, 0);
  // Select high gain first
  driver.Push({GPUExpandEntry{1, 1, low_gain, 2.0f, 1.0f, 1.0f}});
  driver.Push({GPUExpandEntry{2, 2, high_gain, 2.0f, 1.0f, 1.0f}});
  auto res = driver.Pop();
  EXPECT_EQ(res.size(), 1);
  EXPECT_EQ(res[0].nid, 2);
  res = driver.Pop();
  EXPECT_EQ(res.size(), 1);
  EXPECT_EQ(res[0].nid, 1);

  // If equal gain, use nid
  driver.Push({GPUExpandEntry{2, 1, low_gain, 2.0f, 1.0f, 1.0f}});
  driver.Push({GPUExpandEntry{1, 1, low_gain, 2.0f, 1.0f, 1.0f}});
  res = driver.Pop();
  EXPECT_EQ(res[0].nid, 1);
  res = driver.Pop();
  EXPECT_EQ(res[0].nid, 2);
}
}  // namespace tree
}  // namespace xgboost
