#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2024, XGBoost Contributors
 */
#include <gtest/gtest.h>
#include <xgboost/context.h>  // for Context

#include <memory>  // for unique_ptr
#include <vector>  // for vector

#include "../../../../src/tree/gpu_hist/histogram.cuh"
#include "../../../../src/tree/gpu_hist/row_partitioner.cuh"  // for RowPartitioner
#include "../../../../src/tree/param.h"                       // for TrainParam
#include "../../categorical_helpers.h"                        // for OneHotEncodeFeature
#include "../../helpers.h"

namespace xgboost::tree {
TEST(Histogram, DeviceHistogramStorage) {
  // Ensures that node allocates correctly after reaching `kStopGrowingSize`.
  auto ctx = MakeCUDACtx(0);
  constexpr size_t kNBins = 128;
  constexpr int kNNodes = 4;
  constexpr size_t kStopGrowing = kNNodes * kNBins * 2u;
  DeviceHistogramStorage<kStopGrowing> histogram;
  histogram.Init(FstCU(), kNBins);
  for (int i = 0; i < kNNodes; ++i) {
    histogram.AllocateHistograms(&ctx, {i});
  }
  histogram.Reset(&ctx);
  ASSERT_EQ(histogram.Data().size(), kStopGrowing);

  // Use allocated memory but do not erase nidx_map.
  for (int i = 0; i < kNNodes; ++i) {
    histogram.AllocateHistograms(&ctx, {i});
  }
  for (int i = 0; i < kNNodes; ++i) {
    ASSERT_TRUE(histogram.HistogramExists(i));
  }

  // Add two new nodes
  histogram.AllocateHistograms(&ctx, {kNNodes});
  histogram.AllocateHistograms(&ctx, {kNNodes + 1});

  // Old cached nodes should still exist
  for (int i = 0; i < kNNodes; ++i) {
    ASSERT_TRUE(histogram.HistogramExists(i));
  }

  // Should be deleted
  ASSERT_FALSE(histogram.HistogramExists(kNNodes));
  // Most recent node should exist
  ASSERT_TRUE(histogram.HistogramExists(kNNodes + 1));

  // Add same node again - should fail
  EXPECT_ANY_THROW(histogram.AllocateHistograms(&ctx, {kNNodes + 1}););
}

void TestDeterministicHistogram(bool is_dense, int shm_size, bool force_global) {
  Context ctx = MakeCUDACtx(0);
  size_t constexpr kBins = 256, kCols = 120, kRows = 16384, kRounds = 16;
  float constexpr kLower = -1e-2, kUpper = 1e2;

  float sparsity = is_dense ? 0.0f : 0.5f;
  auto matrix = RandomDataGenerator(kRows, kCols, sparsity).GenerateDMatrix();
  auto batch_param = BatchParam{kBins, tree::TrainParam::DftSparseThreshold()};

  for (auto const& batch : matrix->GetBatches<EllpackPage>(&ctx, batch_param)) {
    auto* page = batch.Impl();

    tree::RowPartitioner row_partitioner{&ctx, kRows, page->base_rowid};
    auto ridx = row_partitioner.GetRows(0);

    bst_bin_t num_bins = kBins * kCols;
    dh::device_vector<GradientPairInt64> histogram(num_bins);
    auto d_histogram = dh::ToSpan(histogram);
    auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
    gpair.SetDevice(ctx.Device());

    FeatureGroups feature_groups(page->Cuts(), page->is_dense, shm_size, sizeof(GradientPairInt64));

    auto quantiser = GradientQuantiser(&ctx, gpair.DeviceSpan(), MetaInfo());
    DeviceHistogramBuilder builder;
    builder.Reset(&ctx, feature_groups.DeviceAccessor(ctx.Device()), force_global);
    builder.BuildHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(ctx.Device()),
                           feature_groups.DeviceAccessor(ctx.Device()), gpair.DeviceSpan(), ridx,
                           d_histogram, quantiser);

    std::vector<GradientPairInt64> histogram_h(num_bins);
    dh::safe_cuda(hipMemcpy(histogram_h.data(), d_histogram.data(),
                             num_bins * sizeof(GradientPairInt64), hipMemcpyDeviceToHost));

    for (std::size_t i = 0; i < kRounds; ++i) {
      dh::device_vector<GradientPairInt64> new_histogram(num_bins);
      auto d_new_histogram = dh::ToSpan(new_histogram);

      auto quantiser = GradientQuantiser(&ctx, gpair.DeviceSpan(), MetaInfo());
      DeviceHistogramBuilder builder;
      builder.Reset(&ctx, feature_groups.DeviceAccessor(ctx.Device()), force_global);
      builder.BuildHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(ctx.Device()),
                             feature_groups.DeviceAccessor(ctx.Device()), gpair.DeviceSpan(), ridx,
                             d_new_histogram, quantiser);

      std::vector<GradientPairInt64> new_histogram_h(num_bins);
      dh::safe_cuda(hipMemcpy(new_histogram_h.data(), d_new_histogram.data(),
                               num_bins * sizeof(GradientPairInt64),
                               hipMemcpyDeviceToHost));
      for (size_t j = 0; j < new_histogram_h.size(); ++j) {
        ASSERT_EQ(new_histogram_h[j].GetQuantisedGrad(), histogram_h[j].GetQuantisedGrad());
        ASSERT_EQ(new_histogram_h[j].GetQuantisedHess(), histogram_h[j].GetQuantisedHess());
      }
    }

    {
      auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
      gpair.SetDevice(ctx.Device());

      // Use a single feature group to compute the baseline.
      FeatureGroups single_group(page->Cuts());

      dh::device_vector<GradientPairInt64> baseline(num_bins);
      DeviceHistogramBuilder builder;
      builder.Reset(&ctx, single_group.DeviceAccessor(ctx.Device()), force_global);
      builder.BuildHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(ctx.Device()),
                             single_group.DeviceAccessor(ctx.Device()), gpair.DeviceSpan(), ridx,
                             dh::ToSpan(baseline), quantiser);

      std::vector<GradientPairInt64> baseline_h(num_bins);
      dh::safe_cuda(hipMemcpy(baseline_h.data(), baseline.data().get(),
                               num_bins * sizeof(GradientPairInt64),
                               hipMemcpyDeviceToHost));

      for (size_t i = 0; i < baseline.size(); ++i) {
        EXPECT_NEAR(baseline_h[i].GetQuantisedGrad(), histogram_h[i].GetQuantisedGrad(),
                    baseline_h[i].GetQuantisedGrad() * 1e-3);
      }
    }
  }
}

TEST(Histogram, GPUDeterministic) {
  std::vector<bool> is_dense_array{false, true};
  std::vector<int> shm_sizes{48 * 1024, 64 * 1024, 160 * 1024};
  for (bool is_dense : is_dense_array) {
    for (int shm_size : shm_sizes) {
      for (bool force_global : {true, false}) {
        TestDeterministicHistogram(is_dense, shm_size, force_global);
      }
    }
  }
}

void ValidateCategoricalHistogram(size_t n_categories, common::Span<GradientPairInt64> onehot,
                                  common::Span<GradientPairInt64> cat) {
  auto cat_sum = std::accumulate(cat.cbegin(), cat.cend(), GradientPairInt64{});
  for (size_t c = 0; c < n_categories; ++c) {
    auto zero = onehot[c * 2];
    auto one = onehot[c * 2 + 1];

    auto chosen = cat[c];
    auto not_chosen = cat_sum - chosen;
    ASSERT_EQ(zero, not_chosen);
    ASSERT_EQ(one, chosen);
  }
}

// Test 1 vs rest categorical histogram is equivalent to one hot encoded data.
void TestGPUHistogramCategorical(size_t num_categories) {
  auto ctx = MakeCUDACtx(0);
  size_t constexpr kRows = 340;
  size_t constexpr kBins = 256;
  auto x = GenerateRandomCategoricalSingleColumn(kRows, num_categories);
  auto cat_m = GetDMatrixFromData(x, kRows, 1);
  cat_m->Info().feature_types.HostVector().push_back(FeatureType::kCategorical);
  auto batch_param = BatchParam{kBins, tree::TrainParam::DftSparseThreshold()};
  tree::RowPartitioner row_partitioner{&ctx, kRows, 0};
  auto ridx = row_partitioner.GetRows(0);
  dh::device_vector<GradientPairInt64> cat_hist(num_categories);
  auto gpair = GenerateRandomGradients(kRows, 0, 2);
  gpair.SetDevice(DeviceOrd::CUDA(0));
  auto quantiser = GradientQuantiser(&ctx, gpair.DeviceSpan(), MetaInfo());
  /**
   * Generate hist with cat data.
   */
  for (auto const &batch : cat_m->GetBatches<EllpackPage>(&ctx, batch_param)) {
    auto* page = batch.Impl();
    FeatureGroups single_group(page->Cuts());
    DeviceHistogramBuilder builder;
    builder.Reset(&ctx, single_group.DeviceAccessor(ctx.Device()), false);
    builder.BuildHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(ctx.Device()),
                           single_group.DeviceAccessor(ctx.Device()), gpair.DeviceSpan(), ridx,
                           dh::ToSpan(cat_hist), quantiser);
  }

  /**
   * Generate hist with one hot encoded data.
   */
  auto x_encoded = OneHotEncodeFeature(x, num_categories);
  auto encode_m = GetDMatrixFromData(x_encoded, kRows, num_categories);
  dh::device_vector<GradientPairInt64> encode_hist(2 * num_categories);
  for (auto const &batch : encode_m->GetBatches<EllpackPage>(&ctx, batch_param)) {
    auto* page = batch.Impl();
    FeatureGroups single_group(page->Cuts());
    DeviceHistogramBuilder builder;
    builder.Reset(&ctx, single_group.DeviceAccessor(ctx.Device()), false);
    builder.BuildHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(ctx.Device()),
                           single_group.DeviceAccessor(ctx.Device()), gpair.DeviceSpan(), ridx,
                           dh::ToSpan(encode_hist), quantiser);
  }

  std::vector<GradientPairInt64> h_cat_hist(cat_hist.size());
  thrust::copy(cat_hist.begin(), cat_hist.end(), h_cat_hist.begin());

  std::vector<GradientPairInt64> h_encode_hist(encode_hist.size());
  thrust::copy(encode_hist.begin(), encode_hist.end(), h_encode_hist.begin());
  ValidateCategoricalHistogram(num_categories,
                               common::Span<GradientPairInt64>{h_encode_hist},
                               common::Span<GradientPairInt64>{h_cat_hist});
}

TEST(Histogram, GPUHistCategorical) {
  for (size_t num_categories = 2; num_categories < 8; ++num_categories) {
    TestGPUHistogramCategorical(num_categories);
  }
}

namespace {
// Atomic add as type cast for test.
XGBOOST_DEV_INLINE int64_t atomicAdd(int64_t *dst, int64_t src) {  // NOLINT
  uint64_t* u_dst = reinterpret_cast<uint64_t*>(dst);
  uint64_t u_src = *reinterpret_cast<uint64_t*>(&src);
  uint64_t ret = ::atomicAdd(u_dst, u_src);
  return *reinterpret_cast<int64_t*>(&ret);
}
}

void TestAtomicAdd() {
  size_t n_elements = 1024;
  dh::device_vector<int64_t> result_a(1, 0);
  auto d_result_a = result_a.data().get();

  dh::device_vector<int64_t> result_b(1, 0);
  auto d_result_b = result_b.data().get();

  /**
   * Test for simple inputs
   */
  std::vector<int64_t> h_inputs(n_elements);
  for (size_t i = 0; i < h_inputs.size(); ++i) {
    h_inputs[i] = (i % 2 == 0) ? i : -i;
  }
  dh::device_vector<int64_t> inputs(h_inputs);
  auto d_inputs = inputs.data().get();

  dh::LaunchN(n_elements, [=] __device__(size_t i) {
    AtomicAdd64As32(d_result_a, d_inputs[i]);
    atomicAdd(d_result_b, d_inputs[i]);
  });
  ASSERT_EQ(result_a[0], result_b[0]);

  /**
   * Test for positive values that don't fit into 32 bit integer.
   */
  thrust::fill(inputs.begin(), inputs.end(),
               (std::numeric_limits<uint32_t>::max() / 2));
  thrust::fill(result_a.begin(), result_a.end(), 0);
  thrust::fill(result_b.begin(), result_b.end(), 0);
  dh::LaunchN(n_elements, [=] __device__(size_t i) {
    AtomicAdd64As32(d_result_a, d_inputs[i]);
    atomicAdd(d_result_b, d_inputs[i]);
  });
  ASSERT_EQ(result_a[0], result_b[0]);
  ASSERT_GT(result_a[0], std::numeric_limits<uint32_t>::max());
  CHECK_EQ(thrust::reduce(inputs.begin(), inputs.end(), int64_t(0)), result_a[0]);

  /**
   * Test for negative values that don't fit into 32 bit integer.
   */
  thrust::fill(inputs.begin(), inputs.end(),
               (std::numeric_limits<int32_t>::min() / 2));
  thrust::fill(result_a.begin(), result_a.end(), 0);
  thrust::fill(result_b.begin(), result_b.end(), 0);
  dh::LaunchN(n_elements, [=] __device__(size_t i) {
    AtomicAdd64As32(d_result_a, d_inputs[i]);
    atomicAdd(d_result_b, d_inputs[i]);
  });
  ASSERT_EQ(result_a[0], result_b[0]);
  ASSERT_LT(result_a[0], std::numeric_limits<int32_t>::min());
  CHECK_EQ(thrust::reduce(inputs.begin(), inputs.end(), int64_t(0)), result_a[0]);
}

TEST(Histogram, AtomicAddInt64) {
  TestAtomicAdd();
}

TEST(Histogram, Quantiser) {
  auto ctx = MakeCUDACtx(0);
  std::size_t n_samples{16};
  HostDeviceVector<GradientPair> gpair(n_samples, GradientPair{1.0, 1.0});
  gpair.SetDevice(ctx.Device());

  auto quantiser = GradientQuantiser(&ctx, gpair.DeviceSpan(), MetaInfo());
  for (auto v : gpair.ConstHostVector()) {
    auto gh = quantiser.ToFloatingPoint(quantiser.ToFixedPoint(v));
    ASSERT_EQ(gh.GetGrad(), 1.0);
    ASSERT_EQ(gh.GetHess(), 1.0);
  }
}
namespace {
class HistogramExternalMemoryTest : public ::testing::TestWithParam<std::tuple<float, bool>> {
 public:
  void Run(float sparsity, bool force_global) {
    bst_idx_t n_samples{512}, n_features{12}, n_batches{3};
    std::vector<std::unique_ptr<RowPartitioner>> partitioners;
    auto p_fmat = RandomDataGenerator{n_samples, n_features, sparsity}
                      .Batches(n_batches)
                      .GenerateSparsePageDMatrix("cache", true);
    bst_bin_t n_bins = 16;
    BatchParam p{n_bins, TrainParam::DftSparseThreshold()};
    auto ctx = MakeCUDACtx(0);

    std::unique_ptr<FeatureGroups> fg;
    dh::device_vector<GradientPairInt64> single_hist;
    dh::device_vector<GradientPairInt64> multi_hist;

    auto gpair = GenerateRandomGradients(n_samples);
    gpair.SetDevice(ctx.Device());
    auto quantiser = GradientQuantiser{&ctx, gpair.ConstDeviceSpan(), p_fmat->Info()};
    std::shared_ptr<common::HistogramCuts> cuts;

    {
      /**
       * Multi page.
       */
      std::int32_t k{0};
      for (auto const& page : p_fmat->GetBatches<EllpackPage>(&ctx, p)) {
        auto impl = page.Impl();
        if (k == 0) {
          // Initialization
          auto d_matrix = impl->GetDeviceAccessor(ctx.Device());
          fg = std::make_unique<FeatureGroups>(impl->Cuts());
          auto init = GradientPairInt64{0, 0};
          multi_hist = decltype(multi_hist)(impl->Cuts().TotalBins(), init);
          single_hist = decltype(single_hist)(impl->Cuts().TotalBins(), init);
          cuts = std::make_shared<common::HistogramCuts>(impl->Cuts());
        }

        partitioners.emplace_back(
            std::make_unique<RowPartitioner>(&ctx, impl->Size(), impl->base_rowid));

        auto ridx = partitioners.at(k)->GetRows(0);
        auto d_histogram = dh::ToSpan(multi_hist);
        DeviceHistogramBuilder builder;
        builder.Reset(&ctx, fg->DeviceAccessor(ctx.Device()), force_global);
        builder.BuildHistogram(ctx.CUDACtx(), impl->GetDeviceAccessor(ctx.Device()),
                               fg->DeviceAccessor(ctx.Device()), gpair.ConstDeviceSpan(), ridx,
                               d_histogram, quantiser);
        ++k;
      }
      ASSERT_EQ(k, n_batches);
    }

    {
      /**
       * Single page.
       */
      RowPartitioner partitioner{&ctx, p_fmat->Info().num_row_, 0};
      SparsePage concat;
      std::vector<float> hess(p_fmat->Info().num_row_, 1.0f);
      for (auto const& page : p_fmat->GetBatches<SparsePage>()) {
        concat.Push(page);
      }
      EllpackPageImpl page{&ctx, cuts, concat, p_fmat->IsDense(), p_fmat->Info().num_col_, {}};
      auto ridx = partitioner.GetRows(0);
      auto d_histogram = dh::ToSpan(single_hist);
      DeviceHistogramBuilder builder;
      builder.Reset(&ctx, fg->DeviceAccessor(ctx.Device()), force_global);
      builder.BuildHistogram(ctx.CUDACtx(), page.GetDeviceAccessor(ctx.Device()),
                             fg->DeviceAccessor(ctx.Device()), gpair.ConstDeviceSpan(), ridx,
                             d_histogram, quantiser);
    }

    std::vector<GradientPairInt64> h_single(single_hist.size());
    thrust::copy(single_hist.begin(), single_hist.end(), h_single.begin());
    std::vector<GradientPairInt64> h_multi(multi_hist.size());
    thrust::copy(multi_hist.begin(), multi_hist.end(), h_multi.begin());

    for (std::size_t i = 0; i < single_hist.size(); ++i) {
      ASSERT_EQ(h_single[i].GetQuantisedGrad(), h_multi[i].GetQuantisedGrad());
      ASSERT_EQ(h_single[i].GetQuantisedHess(), h_multi[i].GetQuantisedHess());
    }
  }
};
}  // namespace

TEST_P(HistogramExternalMemoryTest, ExternalMemory) {
  std::apply(&HistogramExternalMemoryTest::Run, std::tuple_cat(std::make_tuple(this), GetParam()));
}

INSTANTIATE_TEST_SUITE_P(Histogram, HistogramExternalMemoryTest, ::testing::ValuesIn([]() {
                           std::vector<std::tuple<float, bool>> params;
                           for (auto global : {true, false}) {
                             for (auto sparsity : {0.0f, 0.2f, 0.8f}) {
                               params.emplace_back(sparsity, global);
                             }
                           }
                           return params;
                         }()));
}  // namespace xgboost::tree
