#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <vector>

#include "../../../../src/common/categorical.h"
#include "../../../../src/tree/gpu_hist/histogram.cuh"
#include "../../../../src/tree/gpu_hist/row_partitioner.cuh"
#include "../../categorical_helpers.h"
#include "../../helpers.h"

namespace xgboost {
namespace tree {

void TestDeterministicHistogram(bool is_dense, int shm_size) {
  Context ctx = CreateEmptyGenericParam(0);
  size_t constexpr kBins = 256, kCols = 120, kRows = 16384, kRounds = 16;
  float constexpr kLower = -1e-2, kUpper = 1e2;

  float sparsity = is_dense ? 0.0f : 0.5f;
  auto matrix = RandomDataGenerator(kRows, kCols, sparsity).GenerateDMatrix();
  BatchParam batch_param{0, static_cast<int32_t>(kBins)};

  for (auto const& batch : matrix->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();

    tree::RowPartitioner row_partitioner(0, kRows);
    auto ridx = row_partitioner.GetRows(0);

    int num_bins = kBins * kCols;
    dh::device_vector<GradientPairInt64> histogram(num_bins);
    auto d_histogram = dh::ToSpan(histogram);
    auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
    gpair.SetDevice(0);

    FeatureGroups feature_groups(page->Cuts(), page->is_dense, shm_size,
                                 sizeof(GradientPairInt64));

    auto quantiser = GradientQuantiser(gpair.DeviceSpan());
    BuildGradientHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(0),
                           feature_groups.DeviceAccessor(0), gpair.DeviceSpan(), ridx, d_histogram,
                           quantiser);

    std::vector<GradientPairInt64> histogram_h(num_bins);
    dh::safe_cuda(hipMemcpy(histogram_h.data(), d_histogram.data(),
                             num_bins * sizeof(GradientPairInt64),
                             hipMemcpyDeviceToHost));

    for (size_t i = 0; i < kRounds; ++i) {
      dh::device_vector<GradientPairInt64> new_histogram(num_bins);
      auto d_new_histogram = dh::ToSpan(new_histogram);

      auto quantiser = GradientQuantiser(gpair.DeviceSpan());
      BuildGradientHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(0),
                             feature_groups.DeviceAccessor(0), gpair.DeviceSpan(), ridx,
                             d_new_histogram, quantiser);

      std::vector<GradientPairInt64> new_histogram_h(num_bins);
      dh::safe_cuda(hipMemcpy(new_histogram_h.data(), d_new_histogram.data(),
                               num_bins * sizeof(GradientPairInt64),
                               hipMemcpyDeviceToHost));
      for (size_t j = 0; j < new_histogram_h.size(); ++j) {
        ASSERT_EQ(new_histogram_h[j].GetQuantisedGrad(), histogram_h[j].GetQuantisedGrad());
        ASSERT_EQ(new_histogram_h[j].GetQuantisedHess(), histogram_h[j].GetQuantisedHess());
      }
    }

    {
      auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
      gpair.SetDevice(0);

      // Use a single feature group to compute the baseline.
      FeatureGroups single_group(page->Cuts());

      dh::device_vector<GradientPairInt64> baseline(num_bins);
      BuildGradientHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(0),
                             single_group.DeviceAccessor(0), gpair.DeviceSpan(), ridx,
                             dh::ToSpan(baseline), quantiser);

      std::vector<GradientPairInt64> baseline_h(num_bins);
      dh::safe_cuda(hipMemcpy(baseline_h.data(), baseline.data().get(),
                               num_bins * sizeof(GradientPairInt64),
                               hipMemcpyDeviceToHost));

      for (size_t i = 0; i < baseline.size(); ++i) {
        EXPECT_NEAR(baseline_h[i].GetQuantisedGrad(), histogram_h[i].GetQuantisedGrad(),
                    baseline_h[i].GetQuantisedGrad() * 1e-3);
      }
    }
  }
}

TEST(Histogram, GPUDeterministic) {
  std::vector<bool> is_dense_array{false, true};
  std::vector<int> shm_sizes{48 * 1024, 64 * 1024, 160 * 1024};
  for (bool is_dense : is_dense_array) {
    for (int shm_size : shm_sizes) {
      TestDeterministicHistogram(is_dense, shm_size);
    }
  }
}

void ValidateCategoricalHistogram(size_t n_categories, common::Span<GradientPairInt64> onehot,
                                  common::Span<GradientPairInt64> cat) {
  auto cat_sum = std::accumulate(cat.cbegin(), cat.cend(), GradientPairInt64{});
  for (size_t c = 0; c < n_categories; ++c) {
    auto zero = onehot[c * 2];
    auto one = onehot[c * 2 + 1];

    auto chosen = cat[c];
    auto not_chosen = cat_sum - chosen;
    ASSERT_EQ(zero, not_chosen);
    ASSERT_EQ(one, chosen);
  }
}

// Test 1 vs rest categorical histogram is equivalent to one hot encoded data.
void TestGPUHistogramCategorical(size_t num_categories) {
  auto ctx = CreateEmptyGenericParam(0);
  size_t constexpr kRows = 340;
  size_t constexpr kBins = 256;
  auto x = GenerateRandomCategoricalSingleColumn(kRows, num_categories);
  auto cat_m = GetDMatrixFromData(x, kRows, 1);
  cat_m->Info().feature_types.HostVector().push_back(FeatureType::kCategorical);
  BatchParam batch_param{0, static_cast<int32_t>(kBins)};
  tree::RowPartitioner row_partitioner(0, kRows);
  auto ridx = row_partitioner.GetRows(0);
  dh::device_vector<GradientPairInt64> cat_hist(num_categories);
  auto gpair = GenerateRandomGradients(kRows, 0, 2);
  gpair.SetDevice(0);
  auto quantiser = GradientQuantiser(gpair.DeviceSpan());
  /**
   * Generate hist with cat data.
   */
  for (auto const &batch : cat_m->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();
    FeatureGroups single_group(page->Cuts());
    BuildGradientHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(0),
                           single_group.DeviceAccessor(0), gpair.DeviceSpan(), ridx,
                           dh::ToSpan(cat_hist), quantiser);
  }

  /**
   * Generate hist with one hot encoded data.
   */
  auto x_encoded = OneHotEncodeFeature(x, num_categories);
  auto encode_m = GetDMatrixFromData(x_encoded, kRows, num_categories);
  dh::device_vector<GradientPairInt64> encode_hist(2 * num_categories);
  for (auto const &batch : encode_m->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();
    FeatureGroups single_group(page->Cuts());
    BuildGradientHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(0),
                           single_group.DeviceAccessor(0), gpair.DeviceSpan(), ridx,
                           dh::ToSpan(encode_hist), quantiser);
  }

  std::vector<GradientPairInt64> h_cat_hist(cat_hist.size());
  thrust::copy(cat_hist.begin(), cat_hist.end(), h_cat_hist.begin());

  std::vector<GradientPairInt64> h_encode_hist(encode_hist.size());
  thrust::copy(encode_hist.begin(), encode_hist.end(), h_encode_hist.begin());
  ValidateCategoricalHistogram(num_categories,
                               common::Span<GradientPairInt64>{h_encode_hist},
                               common::Span<GradientPairInt64>{h_cat_hist});
}

TEST(Histogram, GPUHistCategorical) {
  for (size_t num_categories = 2; num_categories < 8; ++num_categories) {
    TestGPUHistogramCategorical(num_categories);
  }
}

namespace {
// Atomic add as type cast for test.
XGBOOST_DEV_INLINE int64_t atomicAdd(int64_t *dst, int64_t src) {  // NOLINT
  uint64_t* u_dst = reinterpret_cast<uint64_t*>(dst);
  uint64_t u_src = *reinterpret_cast<uint64_t*>(&src);
  uint64_t ret = ::atomicAdd(u_dst, u_src);
  return *reinterpret_cast<int64_t*>(&ret);
}
}

void TestAtomicAdd() {
  size_t n_elements = 1024;
  dh::device_vector<int64_t> result_a(1, 0);
  auto d_result_a = result_a.data().get();

  dh::device_vector<int64_t> result_b(1, 0);
  auto d_result_b = result_b.data().get();

  /**
   * Test for simple inputs
   */
  std::vector<int64_t> h_inputs(n_elements);
  for (size_t i = 0; i < h_inputs.size(); ++i) {
    h_inputs[i] = (i % 2 == 0) ? i : -i;
  }
  dh::device_vector<int64_t> inputs(h_inputs);
  auto d_inputs = inputs.data().get();

  dh::LaunchN(n_elements, [=] __device__(size_t i) {
    AtomicAdd64As32(d_result_a, d_inputs[i]);
    atomicAdd(d_result_b, d_inputs[i]);
  });
  ASSERT_EQ(result_a[0], result_b[0]);

  /**
   * Test for positive values that don't fit into 32 bit integer.
   */
  thrust::fill(inputs.begin(), inputs.end(),
               (std::numeric_limits<uint32_t>::max() / 2));
  thrust::fill(result_a.begin(), result_a.end(), 0);
  thrust::fill(result_b.begin(), result_b.end(), 0);
  dh::LaunchN(n_elements, [=] __device__(size_t i) {
    AtomicAdd64As32(d_result_a, d_inputs[i]);
    atomicAdd(d_result_b, d_inputs[i]);
  });
  ASSERT_EQ(result_a[0], result_b[0]);
  ASSERT_GT(result_a[0], std::numeric_limits<uint32_t>::max());
  CHECK_EQ(thrust::reduce(inputs.begin(), inputs.end(), int64_t(0)), result_a[0]);

  /**
   * Test for negative values that don't fit into 32 bit integer.
   */
  thrust::fill(inputs.begin(), inputs.end(),
               (std::numeric_limits<int32_t>::min() / 2));
  thrust::fill(result_a.begin(), result_a.end(), 0);
  thrust::fill(result_b.begin(), result_b.end(), 0);
  dh::LaunchN(n_elements, [=] __device__(size_t i) {
    AtomicAdd64As32(d_result_a, d_inputs[i]);
    atomicAdd(d_result_b, d_inputs[i]);
  });
  ASSERT_EQ(result_a[0], result_b[0]);
  ASSERT_LT(result_a[0], std::numeric_limits<int32_t>::min());
  CHECK_EQ(thrust::reduce(inputs.begin(), inputs.end(), int64_t(0)), result_a[0]);
}

TEST(Histogram, AtomicAddInt64) {
  TestAtomicAdd();
}
}  // namespace tree
}  // namespace xgboost
