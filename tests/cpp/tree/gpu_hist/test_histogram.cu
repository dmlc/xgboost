#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2024, XGBoost Contributors
 */
#include <gtest/gtest.h>
#include <xgboost/context.h>  // for Context

#include <memory>  // for unique_ptr
#include <vector>  // for vector

#include "../../../../src/tree/gpu_hist/histogram.cuh"
#include "../../../../src/tree/gpu_hist/row_partitioner.cuh"  // for RowPartitioner
#include "../../../../src/tree/hist/param.h"                  // for HistMakerTrainParam
#include "../../../../src/tree/param.h"                       // for TrainParam
#include "../../categorical_helpers.h"                        // for OneHotEncodeFeature
#include "../../helpers.h"
#include "../../histogram_helpers.h"  // for BuildEllpackPage

namespace xgboost::tree {
TEST(Histogram, DeviceHistogramStorage) {
  // Ensures that node allocates correctly after reaching `kStopGrowingSize`.
  auto ctx = MakeCUDACtx(0);
  constexpr size_t kNBins = 128;
  constexpr int kNNodes = 4;
  constexpr size_t kStopGrowing = kNNodes * kNBins * 2u;
  DeviceHistogramStorage histogram{};
  histogram.Reset(&ctx, kNBins, kNNodes);
  for (int i = 0; i < kNNodes; ++i) {
    histogram.AllocateHistograms(&ctx, {i});
  }
  ASSERT_EQ(histogram.Data().size(), kStopGrowing);
  histogram.Reset(&ctx, kNBins, kNNodes);

  // Use allocated memory but do not erase nidx_map.
  for (int i = 0; i < kNNodes; ++i) {
    histogram.AllocateHistograms(&ctx, {i});
  }
  for (int i = 0; i < kNNodes; ++i) {
    ASSERT_TRUE(histogram.HistogramExists(i));
  }

  // Add two new nodes
  histogram.AllocateHistograms(&ctx, {kNNodes});
  histogram.AllocateHistograms(&ctx, {kNNodes + 1});

  // Old cached nodes should still exist
  for (int i = 0; i < kNNodes; ++i) {
    ASSERT_TRUE(histogram.HistogramExists(i));
  }

  // Should be deleted
  ASSERT_FALSE(histogram.HistogramExists(kNNodes));
  // Most recent node should exist
  ASSERT_TRUE(histogram.HistogramExists(kNNodes + 1));

  // Add same node again - should fail
  EXPECT_ANY_THROW(histogram.AllocateHistograms(&ctx, {kNNodes + 1}););
}

TEST(Histogram, SubtractionTrack) {
  auto ctx = MakeCUDACtx(0);

  auto page = BuildEllpackPage(&ctx, 64, 4);
  auto cuts = page->CutsShared();
  FeatureGroups fg{*cuts, true, std::numeric_limits<std::size_t>::max()};
  auto fg_acc = fg.DeviceAccessor(ctx.Device());
  auto n_total_bins = cuts->TotalBins();

  // 2 nodes
  auto max_cached_hist_nodes = 2ull;
  DeviceHistogramBuilder histogram;
  histogram.Reset(&ctx, max_cached_hist_nodes, fg_acc, n_total_bins, false);
  histogram.AllocateHistograms(&ctx, {0, 1, 2});
  GPUExpandEntry root;
  root.nid = 0;
  auto need_build = histogram.SubtractHist(&ctx, {root}, {0}, {1});

  std::vector<GPUExpandEntry> candidates(2);
  candidates[0].nid = 1;
  candidates[1].nid = 2;

  need_build = histogram.SubtractHist(&ctx, candidates, {3, 5}, {4, 6});
  ASSERT_EQ(need_build.size(), 2);
  ASSERT_EQ(need_build[0], 4);
  ASSERT_EQ(need_build[1], 6);
}

std::vector<GradientPairPrecise> GetHostHistGpair() {
  // 24 bins, 3 bins for each feature (column).
  std::vector<GradientPairPrecise> hist_gpair = {
    {0.8314f, 0.7147f}, {1.7989f, 3.7312f}, {3.3846f, 3.4598f},
    {2.9277f, 3.5886f}, {1.8429f, 2.4152f}, {1.2443f, 1.9019f},
    {1.6380f, 2.9174f}, {1.5657f, 2.5107f}, {2.8111f, 2.4776f},
    {2.1322f, 3.0651f}, {3.2927f, 3.8540f}, {0.5899f, 0.9866f},
    {1.5185f, 1.6263f}, {2.0686f, 3.1844f}, {2.4278f, 3.0950f},
    {1.5105f, 2.1403f}, {2.6922f, 4.2217f}, {1.8122f, 1.5437f},
    {0.0000f, 0.0000f}, {4.3245f, 5.7955f}, {1.6903f, 2.1103f},
    {2.4012f, 4.4754f}, {3.6136f, 3.4303f}, {0.0000f, 0.0000f}
  };
  return hist_gpair;
}

void TestBuildHist(bool use_shared_memory_histograms) {
  int const kNRows = 16, kNCols = 8;
  auto ctx = MakeCUDACtx(0);

  auto page = BuildEllpackPage(&ctx, kNRows, kNCols);

  xgboost::SimpleLCG gen;
  xgboost::SimpleRealUniformDistribution<bst_float> dist(0.0f, 1.0f);
  HostDeviceVector<GradientPair> gpair(kNRows);
  for (auto& gp : gpair.HostVector()) {
    float grad = dist(&gen);
    float hess = dist(&gen);
    gp = GradientPair{grad, hess};
  }
  gpair.SetDevice(ctx.Device());

  auto row_partitioner = std::make_unique<RowPartitioner>();
  row_partitioner->Reset(&ctx, kNRows, 0);

  auto quantiser = std::make_unique<GradientQuantiser>(&ctx, gpair.ConstDeviceSpan(), MetaInfo());
  auto shm_size = use_shared_memory_histograms ? dh::MaxSharedMemoryOptin(ctx.Ordinal()) : 0;
  FeatureGroups feature_groups(page->Cuts(), page->IsDenseCompressed(), shm_size);

  DeviceHistogramBuilder builder;
  builder.Reset(&ctx, HistMakerTrainParam::CudaDefaultNodes(),
                feature_groups.DeviceAccessor(ctx.Device()), page->Cuts().TotalBins(),
                !use_shared_memory_histograms);
  builder.AllocateHistograms(&ctx, {0});
  builder.BuildHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(&ctx),
                         feature_groups.DeviceAccessor(ctx.Device()), gpair.DeviceSpan(),
                         row_partitioner->GetRows(0), builder.GetNodeHistogram(0), *quantiser);

  auto node_histogram = builder.GetNodeHistogram(0);

  std::vector<GradientPairInt64> h_result(node_histogram.size());
  dh::CopyDeviceSpanToVector(&h_result, node_histogram);

  std::vector<GradientPairPrecise> solution = GetHostHistGpair();
  for (size_t i = 0; i < h_result.size(); ++i) {
    auto result = quantiser->ToFloatingPoint(h_result[i]);
    ASSERT_NEAR(result.GetGrad(), solution[i].GetGrad(), 0.01f);
    ASSERT_NEAR(result.GetHess(), solution[i].GetHess(), 0.01f);
  }
}

TEST(Histogram, BuildHistGlobalMem) {
  TestBuildHist(false);
}

TEST(Histogram, BuildHistSharedMem) {
  TestBuildHist(true);
}

namespace {
void TestDeterministicHistogram(bool is_dense, std::size_t shm_size, bool force_global) {
  Context ctx = MakeCUDACtx(0);
  size_t constexpr kBins = 256, kCols = 120, kRows = 16384, kRounds = 16;
  float constexpr kLower = -1e-2, kUpper = 1e2;

  float sparsity = is_dense ? 0.0f : 0.5f;
  auto matrix = RandomDataGenerator(kRows, kCols, sparsity).GenerateDMatrix();
  auto batch_param = BatchParam{kBins, tree::TrainParam::DftSparseThreshold()};

  for (auto const& batch : matrix->GetBatches<EllpackPage>(&ctx, batch_param)) {
    auto* page = batch.Impl();

    tree::RowPartitioner row_partitioner;
    row_partitioner.Reset(&ctx, kRows, page->base_rowid);
    auto ridx = row_partitioner.GetRows(0);

    bst_bin_t num_bins = kBins * kCols;
    dh::device_vector<GradientPairInt64> histogram(num_bins);
    auto d_histogram = dh::ToSpan(histogram);
    auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
    gpair.SetDevice(ctx.Device());

    FeatureGroups feature_groups{page->Cuts(), page->IsDenseCompressed(), shm_size};

    auto quantiser = GradientQuantiser(&ctx, gpair.DeviceSpan(), MetaInfo());
    DeviceHistogramBuilder builder;
    builder.Reset(&ctx, HistMakerTrainParam::CudaDefaultNodes(),
                  feature_groups.DeviceAccessor(ctx.Device()), num_bins, force_global);
    builder.BuildHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(&ctx),
                           feature_groups.DeviceAccessor(ctx.Device()), gpair.DeviceSpan(), ridx,
                           d_histogram, quantiser);

    std::vector<GradientPairInt64> histogram_h(num_bins);
    dh::safe_cuda(hipMemcpy(histogram_h.data(), d_histogram.data(),
                             num_bins * sizeof(GradientPairInt64), hipMemcpyDeviceToHost));

    for (std::size_t i = 0; i < kRounds; ++i) {
      dh::device_vector<GradientPairInt64> new_histogram(num_bins);
      auto d_new_histogram = dh::ToSpan(new_histogram);

      auto quantiser = GradientQuantiser(&ctx, gpair.DeviceSpan(), MetaInfo());
      DeviceHistogramBuilder builder;
      builder.Reset(&ctx, HistMakerTrainParam::CudaDefaultNodes(),
                    feature_groups.DeviceAccessor(ctx.Device()), num_bins, force_global);
      builder.BuildHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(&ctx),
                             feature_groups.DeviceAccessor(ctx.Device()), gpair.DeviceSpan(), ridx,
                             d_new_histogram, quantiser);

      std::vector<GradientPairInt64> new_histogram_h(num_bins);
      dh::safe_cuda(hipMemcpy(new_histogram_h.data(), d_new_histogram.data(),
                               num_bins * sizeof(GradientPairInt64), hipMemcpyDeviceToHost));
      for (size_t j = 0; j < new_histogram_h.size(); ++j) {
        ASSERT_EQ(new_histogram_h[j].GetQuantisedGrad(), histogram_h[j].GetQuantisedGrad());
        ASSERT_EQ(new_histogram_h[j].GetQuantisedHess(), histogram_h[j].GetQuantisedHess());
      }
    }

    {
      auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
      gpair.SetDevice(ctx.Device());

      // Use a single feature group to compute the baseline.
      FeatureGroups single_group(page->Cuts());

      dh::device_vector<GradientPairInt64> baseline(num_bins);
      DeviceHistogramBuilder builder;
      // Single group must use global memory.
      builder.Reset(&ctx, HistMakerTrainParam::CudaDefaultNodes(),
                    single_group.DeviceAccessor(ctx.Device()), num_bins, /*force_global=*/true);
      builder.BuildHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(&ctx),
                             single_group.DeviceAccessor(ctx.Device()), gpair.DeviceSpan(), ridx,
                             dh::ToSpan(baseline), quantiser);

      std::vector<GradientPairInt64> baseline_h(num_bins);
      dh::safe_cuda(hipMemcpy(baseline_h.data(), baseline.data().get(),
                               num_bins * sizeof(GradientPairInt64), hipMemcpyDeviceToHost));

      for (size_t i = 0; i < baseline.size(); ++i) {
        ASSERT_NEAR(baseline_h[i].GetQuantisedGrad(), histogram_h[i].GetQuantisedGrad(),
                    baseline_h[i].GetQuantisedGrad() * 1e-3);
      }
    }
  }
}

class TestGPUDeterministic : public ::testing::TestWithParam<std::tuple<bool, std::size_t, bool>> {
 protected:
  void Run() {
    auto [is_dense, shm_size, force_global] = this->GetParam();
    if (shm_size > dh::MaxSharedMemoryOptin(0) && !force_global) {
      force_global = true;  // We will have to skip this test to avoid false check in the builder.
    }
    TestDeterministicHistogram(is_dense, shm_size, force_global);
  }
};
}  // anonymous namespace

TEST_P(TestGPUDeterministic, Histogram) { this->Run(); }

INSTANTIATE_TEST_SUITE_P(Histogram, TestGPUDeterministic,
                         ::testing::Combine(::testing::Bool(),
                                            ::testing::Values(48 * 1024, 64 * 1024, 160 * 1024),
                                            ::testing::Bool()));

void ValidateCategoricalHistogram(size_t n_categories, common::Span<GradientPairInt64> onehot,
                                  common::Span<GradientPairInt64> cat) {
  auto cat_sum = std::accumulate(cat.cbegin(), cat.cend(), GradientPairInt64{});
  for (size_t c = 0; c < n_categories; ++c) {
    auto zero = onehot[c * 2];
    auto one = onehot[c * 2 + 1];

    auto chosen = cat[c];
    auto not_chosen = cat_sum - chosen;
    ASSERT_EQ(zero, not_chosen);
    ASSERT_EQ(one, chosen);
  }
}

// Test 1 vs rest categorical histogram is equivalent to one hot encoded data.
void TestGPUHistogramCategorical(size_t num_categories) {
  auto ctx = MakeCUDACtx(0);
  size_t constexpr kRows = 340;
  size_t constexpr kBins = 256;
  auto x = GenerateRandomCategoricalSingleColumn(kRows, num_categories);
  auto cat_m = GetDMatrixFromData(x, kRows, 1);
  cat_m->Info().feature_types.HostVector().push_back(FeatureType::kCategorical);
  auto batch_param = BatchParam{kBins, tree::TrainParam::DftSparseThreshold()};
  tree::RowPartitioner row_partitioner;
  row_partitioner.Reset(&ctx, kRows, 0);
  auto ridx = row_partitioner.GetRows(0);
  dh::device_vector<GradientPairInt64> cat_hist(num_categories);
  auto gpair = GenerateRandomGradients(kRows, 0, 2);
  gpair.SetDevice(DeviceOrd::CUDA(0));
  auto quantiser = GradientQuantiser(&ctx, gpair.DeviceSpan(), MetaInfo());
  /**
   * Generate hist with cat data.
   */
  for (auto const &batch : cat_m->GetBatches<EllpackPage>(&ctx, batch_param)) {
    auto* page = batch.Impl();
    FeatureGroups single_group(page->Cuts());
    DeviceHistogramBuilder builder;
    builder.Reset(&ctx, HistMakerTrainParam::CudaDefaultNodes(),
                  single_group.DeviceAccessor(ctx.Device()), num_categories, false);
    builder.BuildHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(&ctx),
                           single_group.DeviceAccessor(ctx.Device()), gpair.DeviceSpan(), ridx,
                           dh::ToSpan(cat_hist), quantiser);
  }

  /**
   * Generate hist with one hot encoded data.
   */
  auto x_encoded = OneHotEncodeFeature(x, num_categories);
  auto encode_m = GetDMatrixFromData(x_encoded, kRows, num_categories);
  dh::device_vector<GradientPairInt64> encode_hist(2 * num_categories);
  for (auto const &batch : encode_m->GetBatches<EllpackPage>(&ctx, batch_param)) {
    auto* page = batch.Impl();
    FeatureGroups single_group(page->Cuts());
    DeviceHistogramBuilder builder;
    builder.Reset(&ctx, HistMakerTrainParam::CudaDefaultNodes(),
                  single_group.DeviceAccessor(ctx.Device()), encode_hist.size(), false);
    builder.BuildHistogram(ctx.CUDACtx(), page->GetDeviceAccessor(&ctx),
                           single_group.DeviceAccessor(ctx.Device()), gpair.DeviceSpan(), ridx,
                           dh::ToSpan(encode_hist), quantiser);
  }

  std::vector<GradientPairInt64> h_cat_hist(cat_hist.size());
  thrust::copy(cat_hist.begin(), cat_hist.end(), h_cat_hist.begin());

  std::vector<GradientPairInt64> h_encode_hist(encode_hist.size());
  thrust::copy(encode_hist.begin(), encode_hist.end(), h_encode_hist.begin());
  ValidateCategoricalHistogram(num_categories,
                               common::Span<GradientPairInt64>{h_encode_hist},
                               common::Span<GradientPairInt64>{h_cat_hist});
}

TEST(Histogram, GPUHistCategorical) {
  for (size_t num_categories = 2; num_categories < 8; ++num_categories) {
    TestGPUHistogramCategorical(num_categories);
  }
}

namespace {
// Atomic add as type cast for test.
XGBOOST_DEV_INLINE int64_t atomicAdd(int64_t *dst, int64_t src) {  // NOLINT
  uint64_t* u_dst = reinterpret_cast<uint64_t*>(dst);
  uint64_t u_src = *reinterpret_cast<uint64_t*>(&src);
  uint64_t ret = ::atomicAdd(u_dst, u_src);
  return *reinterpret_cast<int64_t*>(&ret);
}
}

void TestAtomicAdd() {
  size_t n_elements = 1024;
  dh::device_vector<int64_t> result_a(1, 0);
  auto d_result_a = result_a.data().get();

  dh::device_vector<int64_t> result_b(1, 0);
  auto d_result_b = result_b.data().get();

  /**
   * Test for simple inputs
   */
  std::vector<int64_t> h_inputs(n_elements);
  for (size_t i = 0; i < h_inputs.size(); ++i) {
    h_inputs[i] = (i % 2 == 0) ? i : -i;
  }
  dh::device_vector<int64_t> inputs(h_inputs);
  auto d_inputs = inputs.data().get();

  dh::LaunchN(n_elements, [=] __device__(size_t i) {
    AtomicAdd64As32(d_result_a, d_inputs[i]);
    atomicAdd(d_result_b, d_inputs[i]);
  });
  ASSERT_EQ(result_a[0], result_b[0]);

  /**
   * Test for positive values that don't fit into 32 bit integer.
   */
  thrust::fill(inputs.begin(), inputs.end(),
               (std::numeric_limits<uint32_t>::max() / 2));
  thrust::fill(result_a.begin(), result_a.end(), 0);
  thrust::fill(result_b.begin(), result_b.end(), 0);
  dh::LaunchN(n_elements, [=] __device__(size_t i) {
    AtomicAdd64As32(d_result_a, d_inputs[i]);
    atomicAdd(d_result_b, d_inputs[i]);
  });
  ASSERT_EQ(result_a[0], result_b[0]);
  ASSERT_GT(result_a[0], std::numeric_limits<uint32_t>::max());
  CHECK_EQ(thrust::reduce(inputs.begin(), inputs.end(), int64_t(0)), result_a[0]);

  /**
   * Test for negative values that don't fit into 32 bit integer.
   */
  thrust::fill(inputs.begin(), inputs.end(),
               (std::numeric_limits<int32_t>::min() / 2));
  thrust::fill(result_a.begin(), result_a.end(), 0);
  thrust::fill(result_b.begin(), result_b.end(), 0);
  dh::LaunchN(n_elements, [=] __device__(size_t i) {
    AtomicAdd64As32(d_result_a, d_inputs[i]);
    atomicAdd(d_result_b, d_inputs[i]);
  });
  ASSERT_EQ(result_a[0], result_b[0]);
  ASSERT_LT(result_a[0], std::numeric_limits<int32_t>::min());
  CHECK_EQ(thrust::reduce(inputs.begin(), inputs.end(), int64_t(0)), result_a[0]);
}

TEST(Histogram, AtomicAddInt64) {
  TestAtomicAdd();
}

TEST(Histogram, Quantiser) {
  auto ctx = MakeCUDACtx(0);
  std::size_t n_samples{16};
  HostDeviceVector<GradientPair> gpair(n_samples, GradientPair{1.0, 1.0});
  gpair.SetDevice(ctx.Device());

  auto quantiser = GradientQuantiser(&ctx, gpair.DeviceSpan(), MetaInfo());
  for (auto v : gpair.ConstHostVector()) {
    auto gh = quantiser.ToFloatingPoint(quantiser.ToFixedPoint(v));
    ASSERT_EQ(gh.GetGrad(), 1.0);
    ASSERT_EQ(gh.GetHess(), 1.0);
  }
}
namespace {
class HistogramExternalMemoryTest : public ::testing::TestWithParam<std::tuple<float, bool>> {
 public:
  void Run(float sparsity, bool force_global) {
    bst_idx_t n_samples{512}, n_features{12}, n_batches{3};
    std::vector<std::unique_ptr<RowPartitioner>> partitioners;
    auto p_fmat = RandomDataGenerator{n_samples, n_features, sparsity}
                      .Batches(n_batches)
                      .GenerateSparsePageDMatrix("cache", true);
    bst_bin_t n_bins = 16;
    BatchParam p{n_bins, TrainParam::DftSparseThreshold()};
    auto ctx = MakeCUDACtx(0);

    std::unique_ptr<FeatureGroups> fg;
    dh::device_vector<GradientPairInt64> single_hist;
    dh::device_vector<GradientPairInt64> multi_hist;

    auto gpair = GenerateRandomGradients(n_samples);
    gpair.SetDevice(ctx.Device());
    auto quantiser = GradientQuantiser{&ctx, gpair.ConstDeviceSpan(), p_fmat->Info()};
    std::shared_ptr<common::HistogramCuts> cuts;

    {
      /**
       * Multi page.
       */
      std::int32_t k{0};
      for (auto const& page : p_fmat->GetBatches<EllpackPage>(&ctx, p)) {
        auto impl = page.Impl();
        if (k == 0) {
          // Initialization
          fg = std::make_unique<FeatureGroups>(impl->Cuts());
          auto init = GradientPairInt64{0, 0};
          multi_hist = decltype(multi_hist)(impl->Cuts().TotalBins(), init);
          single_hist = decltype(single_hist)(impl->Cuts().TotalBins(), init);
          cuts = std::make_shared<common::HistogramCuts>(impl->Cuts());
        }

        partitioners.emplace_back(std::make_unique<RowPartitioner>());
        partitioners.back()->Reset(&ctx, impl->Size(), impl->base_rowid);

        auto ridx = partitioners.at(k)->GetRows(0);
        auto d_histogram = dh::ToSpan(multi_hist);
        DeviceHistogramBuilder builder;
        builder.Reset(&ctx, HistMakerTrainParam::CudaDefaultNodes(),
                      fg->DeviceAccessor(ctx.Device()), d_histogram.size(), force_global);
        builder.BuildHistogram(ctx.CUDACtx(), impl->GetDeviceAccessor(&ctx),
                               fg->DeviceAccessor(ctx.Device()), gpair.ConstDeviceSpan(), ridx,
                               d_histogram, quantiser);
        ++k;
      }
      ASSERT_EQ(k, n_batches);
    }

    {
      /**
       * Single page.
       */
      RowPartitioner partitioner;
      partitioner.Reset(&ctx, p_fmat->Info().num_row_, 0);

      SparsePage concat;
      std::vector<float> hess(p_fmat->Info().num_row_, 1.0f);
      for (auto const& page : p_fmat->GetBatches<SparsePage>()) {
        concat.Push(page);
      }
      EllpackPageImpl page{&ctx, cuts, concat, p_fmat->IsDense(), p_fmat->Info().num_col_, {}};
      auto ridx = partitioner.GetRows(0);
      auto d_histogram = dh::ToSpan(single_hist);
      DeviceHistogramBuilder builder;
      builder.Reset(&ctx, HistMakerTrainParam::CudaDefaultNodes(), fg->DeviceAccessor(ctx.Device()),
                    d_histogram.size(), force_global);
      builder.BuildHistogram(ctx.CUDACtx(), page.GetDeviceAccessor(&ctx),
                             fg->DeviceAccessor(ctx.Device()), gpair.ConstDeviceSpan(), ridx,
                             d_histogram, quantiser);
    }

    std::vector<GradientPairInt64> h_single(single_hist.size());
    thrust::copy(single_hist.begin(), single_hist.end(), h_single.begin());
    std::vector<GradientPairInt64> h_multi(multi_hist.size());
    thrust::copy(multi_hist.begin(), multi_hist.end(), h_multi.begin());

    for (std::size_t i = 0; i < single_hist.size(); ++i) {
      ASSERT_EQ(h_single[i].GetQuantisedGrad(), h_multi[i].GetQuantisedGrad());
      ASSERT_EQ(h_single[i].GetQuantisedHess(), h_multi[i].GetQuantisedHess());
    }
  }
};
}  // namespace

TEST_P(HistogramExternalMemoryTest, ExternalMemory) {
  std::apply(&HistogramExternalMemoryTest::Run, std::tuple_cat(std::make_tuple(this), GetParam()));
}

INSTANTIATE_TEST_SUITE_P(Histogram, HistogramExternalMemoryTest,
                         ::testing::Combine(::testing::Values(0.0f, 0.2f, 0.8f),
                                            ::testing::Bool()));
}  // namespace xgboost::tree
