#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <vector>
#include <thrust/shuffle.h>
#include <thrust/random.h>

#include "../../../../src/common/categorical.h"
#include "../../../../src/tree/gpu_hist/histogram.cuh"
#include "../../../../src/tree/gpu_hist/row_partitioner.cuh"
#include "../../categorical_helpers.h"
#include "../../helpers.h"

namespace xgboost {
namespace tree {

template <typename Gradient>
void TestDeterministicHistogram(bool is_dense, int shm_size) {
  size_t constexpr kBins = 256, kCols = 120, kRows = 16384, kRounds = 16;
  float constexpr kLower = -1e-2, kUpper = 1e2;

  float sparsity = is_dense ? 0.0f : 0.5f;
  auto matrix = RandomDataGenerator(kRows, kCols, sparsity).GenerateDMatrix();
  BatchParam batch_param{0, static_cast<int32_t>(kBins)};

  for (auto const& batch : matrix->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();

    tree::RowPartitioner row_partitioner(0, kRows);
    auto ridx = row_partitioner.GetRows(0);

    int num_bins = kBins * kCols;
    dh::device_vector<Gradient> histogram(num_bins);
    auto d_histogram = dh::ToSpan(histogram);
    auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
    gpair.SetDevice(0);

    FeatureGroups feature_groups(page->Cuts(), page->is_dense, shm_size,
                                 sizeof(Gradient));

    auto rounding = CreateRoundingFactor<Gradient>(gpair.DeviceSpan());
    BuildGradientHistogram(page->GetDeviceAccessor(0),
                           feature_groups.DeviceAccessor(0), gpair.DeviceSpan(),
                           ridx, d_histogram, rounding);

    std::vector<Gradient> histogram_h(num_bins);
    dh::safe_cuda(hipMemcpy(histogram_h.data(), d_histogram.data(),
                             num_bins * sizeof(Gradient),
                             hipMemcpyDeviceToHost));

    for (size_t i = 0; i < kRounds; ++i) {
      dh::device_vector<Gradient> new_histogram(num_bins);
      auto d_new_histogram = dh::ToSpan(new_histogram);

      auto rounding = CreateRoundingFactor<Gradient>(gpair.DeviceSpan());
      BuildGradientHistogram(page->GetDeviceAccessor(0),
                             feature_groups.DeviceAccessor(0),
                             gpair.DeviceSpan(), ridx, d_new_histogram,
                             rounding);

      std::vector<Gradient> new_histogram_h(num_bins);
      dh::safe_cuda(hipMemcpy(new_histogram_h.data(), d_new_histogram.data(),
                               num_bins * sizeof(Gradient),
                               hipMemcpyDeviceToHost));
      for (size_t j = 0; j < new_histogram_h.size(); ++j) {
        ASSERT_EQ(new_histogram_h[j].GetGrad(), histogram_h[j].GetGrad());
        ASSERT_EQ(new_histogram_h[j].GetHess(), histogram_h[j].GetHess());
      }
    }

    {
      auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
      gpair.SetDevice(0);

      // Use a single feature group to compute the baseline.
      FeatureGroups single_group(page->Cuts());

      dh::device_vector<Gradient> baseline(num_bins);
      BuildGradientHistogram(page->GetDeviceAccessor(0),
                             single_group.DeviceAccessor(0),
                             gpair.DeviceSpan(), ridx, dh::ToSpan(baseline),
                             rounding);

      std::vector<Gradient> baseline_h(num_bins);
      dh::safe_cuda(hipMemcpy(baseline_h.data(), baseline.data().get(),
                               num_bins * sizeof(Gradient),
                               hipMemcpyDeviceToHost));

      for (size_t i = 0; i < baseline.size(); ++i) {
        EXPECT_NEAR(baseline_h[i].GetGrad(), histogram_h[i].GetGrad(),
                    baseline_h[i].GetGrad() * 1e-3);
      }
    }
  }
}

TEST(Histogram, GPUDeterministic) {
  std::vector<bool> is_dense_array{false, true};
  std::vector<int> shm_sizes{48 * 1024, 64 * 1024, 160 * 1024};
  for (bool is_dense : is_dense_array) {
    for (int shm_size : shm_sizes) {
      TestDeterministicHistogram<GradientPairPrecise>(is_dense, shm_size);
    }
  }
}

// Test 1 vs rest categorical histogram is equivalent to one hot encoded data.
void TestGPUHistogramCategorical(size_t num_categories) {
  size_t constexpr kRows = 340;
  size_t constexpr kBins = 256;
  auto x = GenerateRandomCategoricalSingleColumn(kRows, num_categories);
  auto cat_m = GetDMatrixFromData(x, kRows, 1);
  cat_m->Info().feature_types.HostVector().push_back(FeatureType::kCategorical);
  BatchParam batch_param{0, static_cast<int32_t>(kBins)};
  tree::RowPartitioner row_partitioner(0, kRows);
  auto ridx = row_partitioner.GetRows(0);
  dh::device_vector<GradientPairPrecise> cat_hist(num_categories);
  auto gpair = GenerateRandomGradients(kRows, 0, 2);
  gpair.SetDevice(0);
  auto rounding = CreateRoundingFactor<GradientPairPrecise>(gpair.DeviceSpan());
  /**
   * Generate hist with cat data.
   */
  for (auto const &batch : cat_m->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();
    FeatureGroups single_group(page->Cuts());
    BuildGradientHistogram(page->GetDeviceAccessor(0),
                           single_group.DeviceAccessor(0),
                           gpair.DeviceSpan(), ridx, dh::ToSpan(cat_hist),
                           rounding);
  }

  /**
   * Generate hist with one hot encoded data.
   */
  auto x_encoded = OneHotEncodeFeature(x, num_categories);
  auto encode_m = GetDMatrixFromData(x_encoded, kRows, num_categories);
  dh::device_vector<GradientPairPrecise> encode_hist(2 * num_categories);
  for (auto const &batch : encode_m->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();
    FeatureGroups single_group(page->Cuts());
    BuildGradientHistogram(page->GetDeviceAccessor(0),
                           single_group.DeviceAccessor(0),
                           gpair.DeviceSpan(), ridx, dh::ToSpan(encode_hist),
                           rounding);
  }

  std::vector<GradientPairPrecise> h_cat_hist(cat_hist.size());
  thrust::copy(cat_hist.begin(), cat_hist.end(), h_cat_hist.begin());

  std::vector<GradientPairPrecise> h_encode_hist(encode_hist.size());
  thrust::copy(encode_hist.begin(), encode_hist.end(), h_encode_hist.begin());
  ValidateCategoricalHistogram(num_categories,
                               common::Span<GradientPairPrecise>{h_encode_hist},
                               common::Span<GradientPairPrecise>{h_cat_hist});
}

TEST(Histogram, GPUHistCategorical) {
  for (size_t num_categories = 2; num_categories < 8; ++num_categories) {
    TestGPUHistogramCategorical(num_categories);
  }
}

void RunBenchmark(std::string branch, std::string name, std::size_t kCols, std::size_t kRows, float sparsity, int max_depth){
  size_t constexpr kBins = 256;
  float constexpr kLower = -1e-2, kUpper = 1e2;
  auto matrix = RandomDataGenerator(kRows, kCols, sparsity).GenerateDMatrix();
  BatchParam batch_param{0, static_cast<int32_t>(kBins)};
  int num_bins = kBins * kCols;
  dh::device_vector<GradientPairPrecise> histogram(num_bins);
  auto d_histogram = dh::ToSpan(histogram);
  auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
  gpair.SetDevice(0);
  auto rounding = CreateRoundingFactor<GradientPairPrecise>(gpair.DeviceSpan());
  thrust::device_vector<uint32_t> ridx(kRows);
  for (auto const& batch : matrix->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();
    FeatureGroups feature_groups(page->Cuts(), page->is_dense, dh::MaxSharedMemoryOptin(0),
                                 sizeof(GradientPairPrecise));
    for (int depth = 1; depth <= max_depth; depth++) {
      std::size_t depth_rows = kRows / (1 << (depth - 1));
      std::cout << branch << ',' << name << ',' << depth << ',' << depth_rows << ',' << kCols
                << ',';
      thrust::shuffle(ridx.begin(), ridx.end(), thrust::default_random_engine(depth));
      auto d_ridx = dh::ToSpan(ridx).subspan(0, depth_rows);
      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      hipEventRecord(start);
      BuildGradientHistogram(page->GetDeviceAccessor(0), feature_groups.DeviceAccessor(0),
                             gpair.DeviceSpan(), d_ridx, d_histogram, rounding);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      float milliseconds = 0;
      hipEventElapsedTime(&milliseconds, start, stop);

      std::cout << milliseconds << ',' << ((depth_rows * kCols) / milliseconds)/1000 << '\n';
    }
  }
}

TEST(Histogram, Benchmark) {
  float sparsity = 0.0f;
  std::string branch("overflow");
  std::cout << "branch,dataset,depth,rows,cols,time(ms),Million elements/s\n";
  int max_depth = 8;
  RunBenchmark(branch, "epsilon", 2000, 500000, sparsity, max_depth);
  RunBenchmark(branch, "higgs", 32, 10000000, sparsity, max_depth);
  RunBenchmark(branch, "airline", 13, 115000000, sparsity, max_depth);
  RunBenchmark(branch, "year", 90, 515345, sparsity, max_depth);
}


void TestAtomicAddWithOverflow() {
  thrust::device_vector<GradientPairPrecise> histogram(2);
  thrust::device_vector<GradientPair> gpair = std::vector<GradientPair>{{1.0, 1.0}, {-0.01, 0.1}, {0.02, 0.1}, {-2.0, 1.0}};
  auto d_gpair = dh::ToSpan(gpair);
  auto rounding = CreateRoundingFactor<GradientPairPrecise>(d_gpair);
  auto d_histogram = histogram.data().get();
  dh::LaunchN(gpair.size(), [=] __device__(int idx) {
    __shared__ char shared[sizeof(GradientPairInt32)];
    auto shared_histogram = reinterpret_cast<GradientPairInt32*>(shared);
    if (idx == 0) {
      shared_histogram[0] = GradientPairInt32();
    }

    // Global memory version
    GradientPairPrecise truncated{
        TruncateWithRoundingFactor<GradientPairPrecise::ValueT>(rounding.rounding.GetGrad(),
                                                         d_gpair[idx].GetGrad()),
        TruncateWithRoundingFactor<GradientPairPrecise::ValueT>(rounding.rounding.GetHess(),
                                                         d_gpair[idx].GetHess()),
    };
    dh::AtomicAddGpair(d_histogram, truncated);

    // Reduced precision shared memory version
    auto adjusted = rounding.ToFixedPoint(d_gpair[idx]);
    AtomicAddGpairWithOverflow(shared_histogram, adjusted, d_histogram + 1, rounding);
    // First thread copies shared back to global
    if (idx == 0) {
      dh::AtomicAddGpair(d_histogram + 1, rounding.ToFloatingPoint(GradientPairInt64{shared_histogram[idx].GetGrad(),shared_histogram[idx].GetHess()}));
    }
  });

  GradientPairPrecise global = histogram[0];
  GradientPairPrecise shared = histogram[1];
  ASSERT_EQ(global.GetGrad(), shared.GetGrad());
  ASSERT_EQ(global.GetHess(), shared.GetHess());
}

TEST(Histogram, AtomicAddWithOverflow) {
TestAtomicAddWithOverflow();
}

}  // namespace tree
}  // namespace xgboost
