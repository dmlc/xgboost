#include <gtest/gtest.h>
#include <vector>

#include "../../../../src/common/categorical.h"
#include "../../../../src/tree/gpu_hist/histogram.cuh"
#include "../../../../src/tree/gpu_hist/row_partitioner.cuh"
#include "../../categorical_helpers.h"
#include "../../helpers.h"

namespace xgboost {
namespace tree {

template <typename Gradient>
void TestDeterministicHistogram(bool is_dense, int shm_size) {
  size_t constexpr kBins = 256, kCols = 120, kRows = 16384, kRounds = 16;
  float constexpr kLower = -1e-2, kUpper = 1e2;

  float sparsity = is_dense ? 0.0f : 0.5f;
  auto matrix = RandomDataGenerator(kRows, kCols, sparsity).GenerateDMatrix();
  BatchParam batch_param{0, static_cast<int32_t>(kBins)};

  for (auto const& batch : matrix->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();

    tree::RowPartitioner row_partitioner(0, kRows);
    auto ridx = row_partitioner.GetRows(0);

    int num_bins = kBins * kCols;
    dh::device_vector<Gradient> histogram(num_bins);
    auto d_histogram = dh::ToSpan(histogram);
    auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
    gpair.SetDevice(0);

    FeatureGroups feature_groups(page->Cuts(), page->is_dense, shm_size,
                                 sizeof(Gradient));

    auto rounding = CreateRoundingFactor<Gradient>(gpair.DeviceSpan());
    BuildGradientHistogram(page->GetDeviceAccessor(0),
                           feature_groups.DeviceAccessor(0), gpair.DeviceSpan(),
                           ridx, d_histogram, rounding);

    std::vector<Gradient> histogram_h(num_bins);
    dh::safe_cuda(hipMemcpy(histogram_h.data(), d_histogram.data(),
                             num_bins * sizeof(Gradient),
                             hipMemcpyDeviceToHost));

    for (size_t i = 0; i < kRounds; ++i) {
      dh::device_vector<Gradient> new_histogram(num_bins);
      auto d_new_histogram = dh::ToSpan(new_histogram);

      auto rounding = CreateRoundingFactor<Gradient>(gpair.DeviceSpan());
      BuildGradientHistogram(page->GetDeviceAccessor(0),
                             feature_groups.DeviceAccessor(0),
                             gpair.DeviceSpan(), ridx, d_new_histogram,
                             rounding);

      std::vector<Gradient> new_histogram_h(num_bins);
      dh::safe_cuda(hipMemcpy(new_histogram_h.data(), d_new_histogram.data(),
                               num_bins * sizeof(Gradient),
                               hipMemcpyDeviceToHost));
      for (size_t j = 0; j < new_histogram_h.size(); ++j) {
        ASSERT_EQ(new_histogram_h[j].GetGrad(), histogram_h[j].GetGrad());
        ASSERT_EQ(new_histogram_h[j].GetHess(), histogram_h[j].GetHess());
      }
    }

    {
      auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
      gpair.SetDevice(0);

      // Use a single feature group to compute the baseline.
      FeatureGroups single_group(page->Cuts());

      dh::device_vector<Gradient> baseline(num_bins);
      BuildGradientHistogram(page->GetDeviceAccessor(0),
                             single_group.DeviceAccessor(0),
                             gpair.DeviceSpan(), ridx, dh::ToSpan(baseline),
                             rounding);

      std::vector<Gradient> baseline_h(num_bins);
      dh::safe_cuda(hipMemcpy(baseline_h.data(), baseline.data().get(),
                               num_bins * sizeof(Gradient),
                               hipMemcpyDeviceToHost));

      for (size_t i = 0; i < baseline.size(); ++i) {
        EXPECT_NEAR(baseline_h[i].GetGrad(), histogram_h[i].GetGrad(),
                    baseline_h[i].GetGrad() * 1e-3);
      }
    }
  }
}

TEST(Histogram, GPUDeterministic) {
  std::vector<bool> is_dense_array{false, true};
  std::vector<int> shm_sizes{48 * 1024, 64 * 1024, 160 * 1024};
  for (bool is_dense : is_dense_array) {
    for (int shm_size : shm_sizes) {
      TestDeterministicHistogram<GradientPairPrecise>(is_dense, shm_size);
    }
  }
}

// Test 1 vs rest categorical histogram is equivalent to one hot encoded data.
void TestGPUHistogramCategorical(size_t num_categories) {
  size_t constexpr kRows = 340;
  size_t constexpr kBins = 256;
  auto x = GenerateRandomCategoricalSingleColumn(kRows, num_categories);
  auto cat_m = GetDMatrixFromData(x, kRows, 1);
  cat_m->Info().feature_types.HostVector().push_back(FeatureType::kCategorical);
  BatchParam batch_param{0, static_cast<int32_t>(kBins)};
  tree::RowPartitioner row_partitioner(0, kRows);
  auto ridx = row_partitioner.GetRows(0);
  dh::device_vector<GradientPairPrecise> cat_hist(num_categories);
  auto gpair = GenerateRandomGradients(kRows, 0, 2);
  gpair.SetDevice(0);
  auto rounding = CreateRoundingFactor<GradientPairPrecise>(gpair.DeviceSpan());
  /**
   * Generate hist with cat data.
   */
  for (auto const &batch : cat_m->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();
    FeatureGroups single_group(page->Cuts());
    BuildGradientHistogram(page->GetDeviceAccessor(0),
                           single_group.DeviceAccessor(0),
                           gpair.DeviceSpan(), ridx, dh::ToSpan(cat_hist),
                           rounding);
  }

  /**
   * Generate hist with one hot encoded data.
   */
  auto x_encoded = OneHotEncodeFeature(x, num_categories);
  auto encode_m = GetDMatrixFromData(x_encoded, kRows, num_categories);
  dh::device_vector<GradientPairPrecise> encode_hist(2 * num_categories);
  for (auto const &batch : encode_m->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();
    FeatureGroups single_group(page->Cuts());
    BuildGradientHistogram(page->GetDeviceAccessor(0),
                           single_group.DeviceAccessor(0),
                           gpair.DeviceSpan(), ridx, dh::ToSpan(encode_hist),
                           rounding);
  }

  std::vector<GradientPairPrecise> h_cat_hist(cat_hist.size());
  thrust::copy(cat_hist.begin(), cat_hist.end(), h_cat_hist.begin());

  std::vector<GradientPairPrecise> h_encode_hist(encode_hist.size());
  thrust::copy(encode_hist.begin(), encode_hist.end(), h_encode_hist.begin());
  ValidateCategoricalHistogram(num_categories,
                               common::Span<GradientPairPrecise>{h_encode_hist},
                               common::Span<GradientPairPrecise>{h_cat_hist});
}

TEST(Histogram, GPUHistCategorical) {
  for (size_t num_categories = 2; num_categories < 8; ++num_categories) {
    TestGPUHistogramCategorical(num_categories);
  }
}
}  // namespace tree
}  // namespace xgboost
