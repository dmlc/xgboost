#include <gtest/gtest.h>
#include <vector>
#include "../../helpers.h"
#include "../../../../src/tree/gpu_hist/row_partitioner.cuh"
#include "../../../../src/tree/gpu_hist/histogram.cuh"

namespace xgboost {
namespace tree {

template <typename Gradient>
void TestDeterministicHistogram(bool is_dense, int shm_size) {
  size_t constexpr kBins = 256, kCols = 120, kRows = 16384, kRounds = 16;
  float constexpr kLower = -1e-2, kUpper = 1e2;

  float sparsity = is_dense ? 0.0f : 0.5f;
  auto matrix = RandomDataGenerator(kRows, kCols, sparsity).GenerateDMatrix();
  BatchParam batch_param{0, static_cast<int32_t>(kBins), 0};

  for (auto const& batch : matrix->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();

    tree::RowPartitioner row_partitioner(0, kRows);
    auto ridx = row_partitioner.GetRows(0);

    int num_bins = kBins * kCols;
    dh::device_vector<Gradient> histogram(num_bins);
    auto d_histogram = dh::ToSpan(histogram);
    auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
    gpair.SetDevice(0);

    FeatureGroups feature_groups(page->Cuts(), page->is_dense, shm_size,
                                 sizeof(Gradient));
    
    auto rounding = CreateRoundingFactor<Gradient>(gpair.DeviceSpan());
    BuildGradientHistogram(page->GetDeviceAccessor(0),
                           feature_groups.DeviceAccessor(0), gpair.DeviceSpan(),
                           ridx, d_histogram, rounding);

    std::vector<Gradient> histogram_h(num_bins);
    dh::safe_cuda(hipMemcpy(histogram_h.data(), d_histogram.data(),
                             num_bins * sizeof(Gradient),
                             hipMemcpyDeviceToHost));

    for (size_t i = 0; i < kRounds; ++i) {
      dh::device_vector<Gradient> new_histogram(num_bins);
      auto d_new_histogram = dh::ToSpan(new_histogram);

      auto rounding = CreateRoundingFactor<Gradient>(gpair.DeviceSpan());
      BuildGradientHistogram(page->GetDeviceAccessor(0),
                             feature_groups.DeviceAccessor(0),
                             gpair.DeviceSpan(), ridx, d_new_histogram,
                             rounding);

      std::vector<Gradient> new_histogram_h(num_bins);
      dh::safe_cuda(hipMemcpy(new_histogram_h.data(), d_new_histogram.data(),
                               num_bins * sizeof(Gradient),
                               hipMemcpyDeviceToHost));
      for (size_t j = 0; j < new_histogram_h.size(); ++j) {
        ASSERT_EQ(new_histogram_h[j].GetGrad(), histogram_h[j].GetGrad());
        ASSERT_EQ(new_histogram_h[j].GetHess(), histogram_h[j].GetHess());
      }
    }

    {
      auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
      gpair.SetDevice(0);

      // Use a single feature group to compute the baseline.
      FeatureGroups single_group(page->Cuts());
      
      dh::device_vector<Gradient> baseline(num_bins);
      BuildGradientHistogram(page->GetDeviceAccessor(0),
                             single_group.DeviceAccessor(0),
                             gpair.DeviceSpan(), ridx, dh::ToSpan(baseline),
                             rounding);

      std::vector<Gradient> baseline_h(num_bins);
      dh::safe_cuda(hipMemcpy(baseline_h.data(), baseline.data().get(),
                               num_bins * sizeof(Gradient),
                               hipMemcpyDeviceToHost));

      for (size_t i = 0; i < baseline.size(); ++i) {
        EXPECT_NEAR(baseline_h[i].GetGrad(), histogram_h[i].GetGrad(),
                    baseline_h[i].GetGrad() * 1e-3);
      }
    }
  }
}

TEST(Histogram, GPUDeterministic) {
  std::vector<bool> is_dense_array{false, true};
  std::vector<int> shm_sizes{48 * 1024, 64 * 1024, 160 * 1024};
  for (bool is_dense : is_dense_array) {
    for (int shm_size : shm_sizes) {
      TestDeterministicHistogram<GradientPair>(is_dense, shm_size);
      TestDeterministicHistogram<GradientPairPrecise>(is_dense, shm_size);
    }
  }
}
}  // namespace tree
}  // namespace xgboost
