#include <gtest/gtest.h>
#include <vector>
#include "../../helpers.h"
#include "../../../../src/common/categorical.h"
#include "../../../../src/tree/gpu_hist/row_partitioner.cuh"
#include "../../../../src/tree/gpu_hist/histogram.cuh"

namespace xgboost {
namespace tree {

template <typename Gradient>
void TestDeterministicHistogram(bool is_dense, int shm_size) {
  size_t constexpr kBins = 256, kCols = 120, kRows = 16384, kRounds = 16;
  float constexpr kLower = -1e-2, kUpper = 1e2;

  float sparsity = is_dense ? 0.0f : 0.5f;
  auto matrix = RandomDataGenerator(kRows, kCols, sparsity).GenerateDMatrix();
  BatchParam batch_param{0, static_cast<int32_t>(kBins), 0};

  for (auto const& batch : matrix->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();

    tree::RowPartitioner row_partitioner(0, kRows);
    auto ridx = row_partitioner.GetRows(0);

    int num_bins = kBins * kCols;
    dh::device_vector<Gradient> histogram(num_bins);
    auto d_histogram = dh::ToSpan(histogram);
    auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
    gpair.SetDevice(0);

    FeatureGroups feature_groups(page->Cuts(), page->is_dense, shm_size,
                                 sizeof(Gradient));

    auto rounding = CreateRoundingFactor<Gradient>(gpair.DeviceSpan());
    BuildGradientHistogram(page->GetDeviceAccessor(0),
                           feature_groups.DeviceAccessor(0), gpair.DeviceSpan(),
                           ridx, d_histogram, rounding);

    std::vector<Gradient> histogram_h(num_bins);
    dh::safe_cuda(hipMemcpy(histogram_h.data(), d_histogram.data(),
                             num_bins * sizeof(Gradient),
                             hipMemcpyDeviceToHost));

    for (size_t i = 0; i < kRounds; ++i) {
      dh::device_vector<Gradient> new_histogram(num_bins);
      auto d_new_histogram = dh::ToSpan(new_histogram);

      auto rounding = CreateRoundingFactor<Gradient>(gpair.DeviceSpan());
      BuildGradientHistogram(page->GetDeviceAccessor(0),
                             feature_groups.DeviceAccessor(0),
                             gpair.DeviceSpan(), ridx, d_new_histogram,
                             rounding);

      std::vector<Gradient> new_histogram_h(num_bins);
      dh::safe_cuda(hipMemcpy(new_histogram_h.data(), d_new_histogram.data(),
                               num_bins * sizeof(Gradient),
                               hipMemcpyDeviceToHost));
      for (size_t j = 0; j < new_histogram_h.size(); ++j) {
        ASSERT_EQ(new_histogram_h[j].GetGrad(), histogram_h[j].GetGrad());
        ASSERT_EQ(new_histogram_h[j].GetHess(), histogram_h[j].GetHess());
      }
    }

    {
      auto gpair = GenerateRandomGradients(kRows, kLower, kUpper);
      gpair.SetDevice(0);

      // Use a single feature group to compute the baseline.
      FeatureGroups single_group(page->Cuts());

      dh::device_vector<Gradient> baseline(num_bins);
      BuildGradientHistogram(page->GetDeviceAccessor(0),
                             single_group.DeviceAccessor(0),
                             gpair.DeviceSpan(), ridx, dh::ToSpan(baseline),
                             rounding);

      std::vector<Gradient> baseline_h(num_bins);
      dh::safe_cuda(hipMemcpy(baseline_h.data(), baseline.data().get(),
                               num_bins * sizeof(Gradient),
                               hipMemcpyDeviceToHost));

      for (size_t i = 0; i < baseline.size(); ++i) {
        EXPECT_NEAR(baseline_h[i].GetGrad(), histogram_h[i].GetGrad(),
                    baseline_h[i].GetGrad() * 1e-3);
      }
    }
  }
}

TEST(Histogram, GPUDeterministic) {
  std::vector<bool> is_dense_array{false, true};
  std::vector<int> shm_sizes{48 * 1024, 64 * 1024, 160 * 1024};
  for (bool is_dense : is_dense_array) {
    for (int shm_size : shm_sizes) {
      TestDeterministicHistogram<GradientPair>(is_dense, shm_size);
      TestDeterministicHistogram<GradientPairPrecise>(is_dense, shm_size);
    }
  }
}

std::vector<float> OneHotEncodeFeature(std::vector<float> x, size_t num_cat) {
  std::vector<float> ret(x.size() * num_cat, 0);
  size_t n_rows = x.size();
  for (size_t r = 0; r < n_rows; ++r) {
    bst_cat_t cat = common::AsCat(x[r]);
    ret.at(num_cat * r + cat) = 1;
  }
  return ret;
}

// Test 1 vs rest categorical histogram is equivalent to one hot encoded data.
void TestGPUHistogramCategorical(size_t num_categories) {
  size_t constexpr kRows = 340;
  size_t constexpr kBins = 256;
  auto x = GenerateRandomCategoricalSingleColumn(kRows, num_categories);
  auto cat_m = GetDMatrixFromData(x, kRows, 1);
  cat_m->Info().feature_types.HostVector().push_back(FeatureType::kCategorical);
  BatchParam batch_param{0, static_cast<int32_t>(kBins), 0};
  tree::RowPartitioner row_partitioner(0, kRows);
  auto ridx = row_partitioner.GetRows(0);
  dh::device_vector<GradientPairPrecise> cat_hist(num_categories);
  auto gpair = GenerateRandomGradients(kRows, 0, 2);
  gpair.SetDevice(0);
  auto rounding = CreateRoundingFactor<GradientPairPrecise>(gpair.DeviceSpan());
  // Generate hist with cat data.
  for (auto const &batch : cat_m->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();
    FeatureGroups single_group(page->Cuts());
    BuildGradientHistogram(page->GetDeviceAccessor(0),
                           single_group.DeviceAccessor(0),
                           gpair.DeviceSpan(), ridx, dh::ToSpan(cat_hist),
                           rounding);
  }

  // Generate hist with one hot encoded data.
  auto x_encoded = OneHotEncodeFeature(x, num_categories);
  auto encode_m = GetDMatrixFromData(x_encoded, kRows, num_categories);
  dh::device_vector<GradientPairPrecise> encode_hist(2 * num_categories);
  for (auto const &batch : encode_m->GetBatches<EllpackPage>(batch_param)) {
    auto* page = batch.Impl();
    FeatureGroups single_group(page->Cuts());
    BuildGradientHistogram(page->GetDeviceAccessor(0),
                           single_group.DeviceAccessor(0),
                           gpair.DeviceSpan(), ridx, dh::ToSpan(encode_hist),
                           rounding);
  }

  std::vector<GradientPairPrecise> h_cat_hist(cat_hist.size());
  thrust::copy(cat_hist.begin(), cat_hist.end(), h_cat_hist.begin());
  auto cat_sum = std::accumulate(h_cat_hist.begin(), h_cat_hist.end(), GradientPairPrecise{});

  std::vector<GradientPairPrecise> h_encode_hist(encode_hist.size());
  thrust::copy(encode_hist.begin(), encode_hist.end(), h_encode_hist.begin());

  for (size_t c = 0; c < num_categories; ++c) {
    auto zero = h_encode_hist[c * 2];
    auto one = h_encode_hist[c * 2 + 1];

    auto chosen = h_cat_hist[c];
    auto not_chosen = cat_sum - chosen;

    ASSERT_LE(RelError(zero.GetGrad(), not_chosen.GetGrad()), kRtEps);
    ASSERT_LE(RelError(zero.GetHess(), not_chosen.GetHess()), kRtEps);

    ASSERT_LE(RelError(one.GetGrad(), chosen.GetGrad()), kRtEps);
    ASSERT_LE(RelError(one.GetHess(), chosen.GetHess()), kRtEps);
  }
}

TEST(Histogram, GPUHistCategorical) {
  for (size_t num_categories = 2; num_categories < 8; ++num_categories) {
    TestGPUHistogramCategorical(num_categories);
  }
}
}  // namespace tree
}  // namespace xgboost
