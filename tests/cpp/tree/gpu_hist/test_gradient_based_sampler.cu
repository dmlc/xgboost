#include <gtest/gtest.h>

#include "../../../../src/data/ellpack_page.cuh"
#include "../../../../src/tree/gpu_hist/gradient_based_sampler.cuh"
#include "../../helpers.h"
#include "dmlc/filesystem.h"

namespace xgboost {
namespace tree {

void VerifySampling(size_t page_size,
                    float subsample,
                    int sampling_method,
                    bool fixed_size_sampling = true,
                    bool check_sum = true) {
  constexpr size_t kRows = 4096;
  constexpr size_t kCols = 1;
  size_t sample_rows = kRows * subsample;

  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix> dmat(
      CreateSparsePageDMatrixWithRC(kRows, kCols, page_size, true, tmpdir));
  auto gpair = GenerateRandomGradients(kRows);
  GradientPair sum_gpair{};
  for (const auto& gp : gpair.ConstHostVector()) {
    sum_gpair += gp;
  }
  gpair.SetDevice(0);

  BatchParam param{0, 256, page_size};
  auto page = (*dmat->GetBatches<EllpackPage>(param).begin()).Impl();
  if (page_size != 0) {
    EXPECT_NE(page->n_rows, kRows);
  }

  GradientBasedSampler sampler(page, kRows, param, subsample, sampling_method);
  auto sample = sampler.Sample(gpair.DeviceSpan(), dmat.get());

  if (fixed_size_sampling) {
    EXPECT_EQ(sample.sample_rows, kRows);
    EXPECT_EQ(sample.page->n_rows, kRows);
    EXPECT_EQ(sample.gpair.size(), kRows);
  } else {
    EXPECT_NEAR(sample.sample_rows, sample_rows, kRows * 0.03);
    EXPECT_NEAR(sample.page->n_rows, sample_rows, kRows * 0.03f);
    EXPECT_NEAR(sample.gpair.size(), sample_rows, kRows * 0.03f);
  }

  GradientPair sum_sampled_gpair{};
  std::vector<GradientPair> sampled_gpair_h(sample.gpair.size());
  dh::CopyDeviceSpanToVector(&sampled_gpair_h, sample.gpair);
  for (const auto& gp : sampled_gpair_h) {
    sum_sampled_gpair += gp;
  }
  if (check_sum) {
    EXPECT_NEAR(sum_gpair.GetGrad(), sum_sampled_gpair.GetGrad(), 0.03f * kRows);
    EXPECT_NEAR(sum_gpair.GetHess(), sum_sampled_gpair.GetHess(), 0.03f * kRows);
  } else {
    EXPECT_NEAR(sum_gpair.GetGrad() / kRows, sum_sampled_gpair.GetGrad() / sample_rows, 0.03f);
    EXPECT_NEAR(sum_gpair.GetHess() / kRows, sum_sampled_gpair.GetHess() / sample_rows, 0.03f);
  }
}

TEST(GradientBasedSampler, NoSampling) {
  constexpr size_t kPageSize = 0;
  constexpr float kSubsample = 1.0f;
  constexpr int kSamplingMethod = TrainParam::kUniform;
  VerifySampling(kPageSize, kSubsample, kSamplingMethod);
}

// In external mode, when not sampling, we concatenate the pages together.
TEST(GradientBasedSampler, NoSamplingExternalMemory) {
  constexpr size_t kRows = 2048;
  constexpr size_t kCols = 1;
  constexpr float kSubsample = 1.0f;
  constexpr size_t kPageSize = 1024;

  // Create a DMatrix with multiple batches.
  dmlc::TemporaryDirectory tmpdir;
  std::unique_ptr<DMatrix>
      dmat(CreateSparsePageDMatrixWithRC(kRows, kCols, kPageSize, true, tmpdir));
  auto gpair = GenerateRandomGradients(kRows);
  gpair.SetDevice(0);

  BatchParam param{0, 256, kPageSize};
  auto page = (*dmat->GetBatches<EllpackPage>(param).begin()).Impl();
  EXPECT_NE(page->n_rows, kRows);

  GradientBasedSampler sampler(page, kRows, param, kSubsample, TrainParam::kUniform);
  auto sample = sampler.Sample(gpair.DeviceSpan(), dmat.get());
  auto sampled_page = sample.page;
  EXPECT_EQ(sample.sample_rows, kRows);
  EXPECT_EQ(sample.gpair.size(), gpair.Size());
  EXPECT_EQ(sample.gpair.data(), gpair.DevicePointer());
  EXPECT_EQ(sampled_page->n_rows, kRows);

  std::vector<common::CompressedByteT> buffer(sampled_page->gidx_buffer.HostVector());
  common::CompressedIterator<common::CompressedByteT>
      ci(buffer.data(), sampled_page->NumSymbols());

  size_t offset = 0;
  for (auto& batch : dmat->GetBatches<EllpackPage>(param)) {
    auto page = batch.Impl();
    std::vector<common::CompressedByteT> page_buffer(page->gidx_buffer.HostVector());
    common::CompressedIterator<common::CompressedByteT>
        page_ci(page_buffer.data(), page->NumSymbols());
    size_t num_elements = page->n_rows * page->row_stride;
    for (size_t i = 0; i < num_elements; i++) {
      EXPECT_EQ(ci[i + offset], page_ci[i]);
    }
    offset += num_elements;
  }
}

TEST(GradientBasedSampler, UniformSampling) {
  constexpr size_t kPageSize = 0;
  constexpr float kSubsample = 0.5;
  constexpr int kSamplingMethod = TrainParam::kUniform;
  constexpr bool kFixedSizeSampling = true;
  constexpr bool kCheckSum = false;
  VerifySampling(kPageSize, kSubsample, kSamplingMethod, kFixedSizeSampling, kCheckSum);
}

TEST(GradientBasedSampler, UniformSamplingExternalMemory) {
  constexpr size_t kPageSize = 1024;
  constexpr float kSubsample = 0.5;
  constexpr int kSamplingMethod = TrainParam::kUniform;
  constexpr bool kFixedSizeSampling = false;
  constexpr bool kCheckSum = false;
  VerifySampling(kPageSize, kSubsample, kSamplingMethod, kFixedSizeSampling, kCheckSum);
}

TEST(GradientBasedSampler, GradientBasedSampling) {
  constexpr size_t kPageSize = 0;
  constexpr float kSubsample = 0.8;
  constexpr int kSamplingMethod = TrainParam::kGradientBased;
  VerifySampling(kPageSize, kSubsample, kSamplingMethod);
}

TEST(GradientBasedSampler, GradientBasedSamplingExternalMemory) {
  constexpr size_t kPageSize = 1024;
  constexpr float kSubsample = 0.8;
  constexpr int kSamplingMethod = TrainParam::kGradientBased;
  constexpr bool kFixedSizeSampling = false;
  VerifySampling(kPageSize, kSubsample, kSamplingMethod, kFixedSizeSampling);
}

};  // namespace tree
};  // namespace xgboost
