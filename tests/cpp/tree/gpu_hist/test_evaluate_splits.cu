#include "hip/hip_runtime.h"
/*!
 * Copyright 2020-2022 by XGBoost contributors
 */
#include <gtest/gtest.h>

#include "../../../../src/tree/gpu_hist/evaluate_splits.cuh"
#include "../../helpers.h"
#include "../../histogram_helpers.h"
#include "../test_evaluate_splits.h"  // TestPartitionBasedSplit

namespace xgboost {
namespace tree {
namespace {
auto ZeroParam() {
  auto args = Args{{"min_child_weight", "0"},
                   {"lambda", "0"}};
  TrainParam tparam;
  tparam.UpdateAllowUnknown(args);
  return tparam;
}

}  // anonymous namespace

TEST_F(TestCategoricalSplitWithMissing, GPUHistEvaluator) {
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0};
  GPUTrainingParam param{param_};
  cuts_.cut_ptrs_.SetDevice(0);
  cuts_.cut_values_.SetDevice(0);
  cuts_.min_vals_.SetDevice(0);
  thrust::device_vector<GradientPairPrecise> feature_histogram{feature_histogram_};

  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  auto d_feature_types = dh::ToSpan(feature_types);

  EvaluateSplitInputs input{1, 0, parent_sum_, dh::ToSpan(feature_set),
                            dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{
      param,
      d_feature_types,
      cuts_.cut_ptrs_.ConstDeviceSpan(),
      cuts_.cut_values_.ConstDeviceSpan(),
      cuts_.min_vals_.ConstDeviceSpan(),
  };

  GPUHistEvaluator evaluator{param_, static_cast<bst_feature_t>(feature_set.size()), 0};

  evaluator.Reset(cuts_, dh::ToSpan(feature_types), feature_set.size(), param_, 0);
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;

  ASSERT_EQ(result.thresh, 1);
  this->CheckResult(result.loss_chg, result.findex, result.fvalue, result.is_cat,
                    result.dir == kLeftDir, result.left_sum, result.right_sum);
}

TEST(GpuHist, PartitionBasic) {
  TrainParam tparam = ZeroParam();
  tparam.max_cat_to_onehot = 0;
  GPUTrainingParam param{tparam};

  common::HistogramCuts cuts;
  cuts.cut_values_.HostVector() = std::vector<float>{0.0, 1.0, 2.0};
  cuts.cut_ptrs_.HostVector() = std::vector<uint32_t>{0, 3};
  cuts.min_vals_.HostVector() = std::vector<float>{0.0};
  cuts.cut_ptrs_.SetDevice(0);
  cuts.cut_values_.SetDevice(0);
  cuts.min_vals_.SetDevice(0);
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0};

  thrust::device_vector<int> monotonic_constraints(feature_set.size(), 0);
  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  common::Span<FeatureType> d_feature_types;
  auto max_cat =
      *std::max_element(cuts.cut_values_.HostVector().begin(), cuts.cut_values_.HostVector().end());
  cuts.SetCategorical(true, max_cat);
  d_feature_types = dh::ToSpan(feature_types);

  EvaluateSplitSharedInputs shared_inputs{
      param,
      d_feature_types,
      cuts.cut_ptrs_.ConstDeviceSpan(),
      cuts.cut_values_.ConstDeviceSpan(),
      cuts.min_vals_.ConstDeviceSpan(),
  };

  GPUHistEvaluator evaluator{tparam, static_cast<bst_feature_t>(feature_set.size()), 0};
  evaluator.Reset(cuts, dh::ToSpan(feature_types), feature_set.size(), tparam, 0);

  {
    // -1.0s go right
    // -3.0s go left
    GradientPairPrecise parent_sum(-5.0, 3.0);
    thrust::device_vector<GradientPairPrecise> feature_histogram =
        std::vector<GradientPairPrecise>{{-1.0, 1.0}, {-1.0, 1.0}, {-3.0, 1.0}};
    EvaluateSplitInputs input{0, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_EQ(cats, std::bitset<32>("11000000000000000000000000000000"));
    EXPECT_FLOAT_EQ(result.left_sum.GetGrad() + result.right_sum.GetGrad(), parent_sum.GetGrad());
    EXPECT_FLOAT_EQ(result.left_sum.GetHess() + result.right_sum.GetHess(), parent_sum.GetHess());
  }

  {
    // -1.0s go right
    // -3.0s go left
    GradientPairPrecise parent_sum(-7.0, 3.0);
    thrust::device_vector<GradientPairPrecise> feature_histogram =
        std::vector<GradientPairPrecise>{{-1.0, 1.0}, {-3.0, 1.0}, {-3.0, 1.0}};
    EvaluateSplitInputs input{1, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_EQ(cats, std::bitset<32>("10000000000000000000000000000000"));
    EXPECT_FLOAT_EQ(result.left_sum.GetGrad() + result.right_sum.GetGrad(), parent_sum.GetGrad());
    EXPECT_FLOAT_EQ(result.left_sum.GetHess() + result.right_sum.GetHess(), parent_sum.GetHess());
  }
  {
    // All -1.0, gain from splitting should be 0.0
    GradientPairPrecise parent_sum(-3.0, 3.0);
    thrust::device_vector<GradientPairPrecise> feature_histogram =
        std::vector<GradientPairPrecise>{{-1.0, 1.0}, {-1.0, 1.0}, {-1.0, 1.0}};
    EvaluateSplitInputs input{2, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_FLOAT_EQ(result.loss_chg, 0.0f);
    EXPECT_FLOAT_EQ(result.left_sum.GetGrad() + result.right_sum.GetGrad(), parent_sum.GetGrad());
    EXPECT_FLOAT_EQ(result.left_sum.GetHess() + result.right_sum.GetHess(), parent_sum.GetHess());
  }
  // With 3.0/3.0 missing values
  // Forward, first 2 categories are selected, while the last one go to left along with missing value
  {
    GradientPairPrecise parent_sum(0.0, 6.0);
    thrust::device_vector<GradientPairPrecise> feature_histogram =
        std::vector<GradientPairPrecise>{{-1.0, 1.0}, {-1.0, 1.0}, {-1.0, 1.0}};
    EvaluateSplitInputs input{3, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(cats, std::bitset<32>("11000000000000000000000000000000"));
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_FLOAT_EQ(result.left_sum.GetGrad() + result.right_sum.GetGrad(), parent_sum.GetGrad());
    EXPECT_FLOAT_EQ(result.left_sum.GetHess() + result.right_sum.GetHess(), parent_sum.GetHess());
  }
  {
    // -1.0s go right
    // -3.0s go left
    GradientPairPrecise parent_sum(-5.0, 3.0);
    thrust::device_vector<GradientPairPrecise> feature_histogram =
        std::vector<GradientPairPrecise>{{-1.0, 1.0}, {-3.0, 1.0}, {-1.0, 1.0}};
    EvaluateSplitInputs input{4, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_EQ(cats, std::bitset<32>("10100000000000000000000000000000"));
    EXPECT_FLOAT_EQ(result.left_sum.GetGrad() + result.right_sum.GetGrad(), parent_sum.GetGrad());
    EXPECT_FLOAT_EQ(result.left_sum.GetHess() + result.right_sum.GetHess(), parent_sum.GetHess());
  }
  {
    // -1.0s go right
    // -3.0s go left
    GradientPairPrecise parent_sum(-5.0, 3.0);
    thrust::device_vector<GradientPairPrecise> feature_histogram =
        std::vector<GradientPairPrecise>{{-3.0, 1.0}, {-1.0, 1.0}, {-3.0, 1.0}};
    EvaluateSplitInputs input{5, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(cats, std::bitset<32>("01000000000000000000000000000000"));
    EXPECT_FLOAT_EQ(result.left_sum.GetGrad() + result.right_sum.GetGrad(), parent_sum.GetGrad());
    EXPECT_FLOAT_EQ(result.left_sum.GetHess() + result.right_sum.GetHess(), parent_sum.GetHess());
  }
}

TEST(GpuHist, PartitionTwoFeatures) {
  TrainParam tparam = ZeroParam();
  tparam.max_cat_to_onehot = 0;
  GPUTrainingParam param{tparam};

  common::HistogramCuts cuts;
  cuts.cut_values_.HostVector() = std::vector<float>{0.0, 1.0, 2.0, 0.0, 1.0, 2.0};
  cuts.cut_ptrs_.HostVector() = std::vector<uint32_t>{0, 3, 6};
  cuts.min_vals_.HostVector() = std::vector<float>{0.0, 0.0};
  cuts.cut_ptrs_.SetDevice(0);
  cuts.cut_values_.SetDevice(0);
  cuts.min_vals_.SetDevice(0);
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0, 1};

  thrust::device_vector<int> monotonic_constraints(feature_set.size(), 0);
  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  common::Span<FeatureType> d_feature_types(dh::ToSpan(feature_types));
  auto max_cat =
      *std::max_element(cuts.cut_values_.HostVector().begin(), cuts.cut_values_.HostVector().end());
  cuts.SetCategorical(true, max_cat);

  EvaluateSplitSharedInputs shared_inputs{
      param,
      d_feature_types,
      cuts.cut_ptrs_.ConstDeviceSpan(),
      cuts.cut_values_.ConstDeviceSpan(),
      cuts.min_vals_.ConstDeviceSpan(),
  };

  GPUHistEvaluator evaluator{tparam, static_cast<bst_feature_t>(feature_set.size()), 0};
  evaluator.Reset(cuts, dh::ToSpan(feature_types), feature_set.size(), tparam, 0);

  {
    GradientPairPrecise parent_sum(-6.0, 3.0);
    thrust::device_vector<GradientPairPrecise> feature_histogram = std::vector<GradientPairPrecise>{
        {-2.0, 1.0}, {-2.0, 1.0}, {-2.0, 1.0}, {-1.0, 1.0}, {-1.0, 1.0}, {-4.0, 1.0}};
    EvaluateSplitInputs input{0, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.findex, 1);
    EXPECT_EQ(cats, std::bitset<32>("11000000000000000000000000000000"));
    EXPECT_FLOAT_EQ(result.left_sum.GetGrad() + result.right_sum.GetGrad(), parent_sum.GetGrad());
    EXPECT_FLOAT_EQ(result.left_sum.GetHess() + result.right_sum.GetHess(), parent_sum.GetHess());
  }

  {
    GradientPairPrecise parent_sum(-6.0, 3.0);
    thrust::device_vector<GradientPairPrecise> feature_histogram = std::vector<GradientPairPrecise>{
        {-2.0, 1.0}, {-2.0, 1.0}, {-2.0, 1.0}, {-1.0, 1.0}, {-2.5, 1.0}, {-2.5, 1.0}};
    EvaluateSplitInputs input{1, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.findex, 1);
    EXPECT_EQ(cats, std::bitset<32>("10000000000000000000000000000000"));
    EXPECT_FLOAT_EQ(result.left_sum.GetGrad() + result.right_sum.GetGrad(), parent_sum.GetGrad());
    EXPECT_FLOAT_EQ(result.left_sum.GetHess() + result.right_sum.GetHess(), parent_sum.GetHess());
  }
}

TEST(GpuHist, PartitionTwoNodes) {
  TrainParam tparam = ZeroParam();
  tparam.max_cat_to_onehot = 0;
  GPUTrainingParam param{tparam};

  common::HistogramCuts cuts;
  cuts.cut_values_.HostVector() = std::vector<float>{0.0, 1.0, 2.0};
  cuts.cut_ptrs_.HostVector() = std::vector<uint32_t>{0, 3};
  cuts.min_vals_.HostVector() = std::vector<float>{0.0};
  cuts.cut_ptrs_.SetDevice(0);
  cuts.cut_values_.SetDevice(0);
  cuts.min_vals_.SetDevice(0);
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0};

  thrust::device_vector<int> monotonic_constraints(feature_set.size(), 0);
  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  common::Span<FeatureType> d_feature_types(dh::ToSpan(feature_types));
  auto max_cat =
      *std::max_element(cuts.cut_values_.HostVector().begin(), cuts.cut_values_.HostVector().end());
  cuts.SetCategorical(true, max_cat);

  EvaluateSplitSharedInputs shared_inputs{
      param,
      d_feature_types,
      cuts.cut_ptrs_.ConstDeviceSpan(),
      cuts.cut_values_.ConstDeviceSpan(),
      cuts.min_vals_.ConstDeviceSpan(),
  };

  GPUHistEvaluator evaluator{tparam, static_cast<bst_feature_t>(feature_set.size()), 0};
  evaluator.Reset(cuts, dh::ToSpan(feature_types), feature_set.size(), tparam, 0);

  {
    GradientPairPrecise parent_sum(-6.0, 3.0);
    thrust::device_vector<GradientPairPrecise> feature_histogram_a =
        std::vector<GradientPairPrecise>{{-1.0, 1.0}, {-2.5, 1.0}, {-2.5, 1.0},
                                         {-1.0, 1.0}, {-1.0, 1.0}, {-4.0, 1.0}};
    thrust::device_vector<EvaluateSplitInputs> inputs(2);
    inputs[0] = EvaluateSplitInputs{0, 0, parent_sum, dh::ToSpan(feature_set),
                                    dh::ToSpan(feature_histogram_a)};
    thrust::device_vector<GradientPairPrecise> feature_histogram_b =
        std::vector<GradientPairPrecise>{{-1.0, 1.0}, {-1.0, 1.0}, {-4.0, 1.0}};
    inputs[1] = EvaluateSplitInputs{1, 0, parent_sum, dh::ToSpan(feature_set),
                                    dh::ToSpan(feature_histogram_b)};
    thrust::device_vector<GPUExpandEntry> results(2);
    evaluator.EvaluateSplits({0, 1}, 1, dh::ToSpan(inputs), shared_inputs, dh::ToSpan(results));
    GPUExpandEntry result_a = results[0];
    GPUExpandEntry result_b = results[1];
    EXPECT_EQ(std::bitset<32>(evaluator.GetHostNodeCats(0)[0]),
              std::bitset<32>("10000000000000000000000000000000"));
    EXPECT_EQ(std::bitset<32>(evaluator.GetHostNodeCats(1)[0]),
              std::bitset<32>("11000000000000000000000000000000"));
  }
}

void TestEvaluateSingleSplit(bool is_categorical) {
  GradientPairPrecise parent_sum(0.0, 1.0);
  TrainParam tparam = ZeroParam();
  GPUTrainingParam param{tparam};

  common::HistogramCuts cuts{MakeCutsForTest({1.0, 2.0, 11.0, 12.0}, {0, 2, 4}, {0.0, 0.0}, 0)};
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0, 1};

  // Setup gradients so that second feature gets higher gain
  thrust::device_vector<GradientPairPrecise> feature_histogram =
      std::vector<GradientPairPrecise>{
          {-0.5, 0.5}, {0.5, 0.5}, {-1.0, 0.5}, {1.0, 0.5}};

  dh::device_vector<FeatureType> feature_types(feature_set.size(),
                                               FeatureType::kCategorical);
  common::Span<FeatureType> d_feature_types;
  if (is_categorical) {
    auto max_cat = *std::max_element(cuts.cut_values_.HostVector().begin(),
                                     cuts.cut_values_.HostVector().end());
    cuts.SetCategorical(true, max_cat);
    d_feature_types = dh::ToSpan(feature_types);
  }

  EvaluateSplitInputs input{1, 0, parent_sum, dh::ToSpan(feature_set),
                            dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{
      param,
      d_feature_types,
      cuts.cut_ptrs_.ConstDeviceSpan(),
      cuts.cut_values_.ConstDeviceSpan(),
      cuts.min_vals_.ConstDeviceSpan(),
  };

  GPUHistEvaluator evaluator{
      tparam, static_cast<bst_feature_t>(feature_set.size()), 0};
  evaluator.Reset(cuts, dh::ToSpan(feature_types), feature_set.size(), tparam, 0);
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;

  EXPECT_EQ(result.findex, 1);
  if (is_categorical) {
    ASSERT_TRUE(std::isnan(result.fvalue));
  } else {
    EXPECT_EQ(result.fvalue, 11.0);
  }
  EXPECT_FLOAT_EQ(result.left_sum.GetGrad() + result.right_sum.GetGrad(),
                  parent_sum.GetGrad());
  EXPECT_FLOAT_EQ(result.left_sum.GetHess() + result.right_sum.GetHess(),
                  parent_sum.GetHess());
}

TEST(GpuHist, EvaluateSingleSplit) {
  TestEvaluateSingleSplit(false);
}

TEST(GpuHist, EvaluateSingleCategoricalSplit) {
  TestEvaluateSingleSplit(true);
}

TEST(GpuHist, EvaluateSingleSplitMissing) {
  GradientPairPrecise parent_sum(1.0, 1.5);
  TrainParam tparam = ZeroParam();
  GPUTrainingParam param{tparam};

  thrust::device_vector<bst_feature_t> feature_set =
      std::vector<bst_feature_t>{0};
  thrust::device_vector<uint32_t> feature_segments =
      std::vector<bst_row_t>{0, 2};
  thrust::device_vector<float> feature_values = std::vector<float>{1.0, 2.0};
  thrust::device_vector<float> feature_min_values = std::vector<float>{0.0};
  thrust::device_vector<GradientPairPrecise> feature_histogram =
      std::vector<GradientPairPrecise>{{-0.5, 0.5}, {0.5, 0.5}};
  EvaluateSplitInputs input{1,0,
                                          parent_sum,
                                          dh::ToSpan(feature_set),
                                          dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{
      param,
      {},
      dh::ToSpan(feature_segments),
      dh::ToSpan(feature_values),
      dh::ToSpan(feature_min_values),
  };

  GPUHistEvaluator evaluator(tparam, feature_set.size(), 0);
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;

  EXPECT_EQ(result.findex, 0);
  EXPECT_EQ(result.fvalue, 1.0);
  EXPECT_EQ(result.dir, kRightDir);
  EXPECT_EQ(result.left_sum, GradientPairPrecise(-0.5, 0.5));
  EXPECT_EQ(result.right_sum, GradientPairPrecise(1.5, 1.0));
}

TEST(GpuHist, EvaluateSingleSplitEmpty) {
  TrainParam tparam = ZeroParam();
  GPUHistEvaluator evaluator(tparam, 1, 0);
  DeviceSplitCandidate result =
      evaluator.EvaluateSingleSplit(EvaluateSplitInputs{}, EvaluateSplitSharedInputs{}).split;
  EXPECT_EQ(result.findex, -1);
  EXPECT_LT(result.loss_chg, 0.0f);
}

// Feature 0 has a better split, but the algorithm must select feature 1
TEST(GpuHist, EvaluateSingleSplitFeatureSampling) {
  GradientPairPrecise parent_sum(0.0, 1.0);
  TrainParam tparam = ZeroParam();
  tparam.UpdateAllowUnknown(Args{});
  GPUTrainingParam param{tparam};

  thrust::device_vector<bst_feature_t> feature_set =
      std::vector<bst_feature_t>{1};
  thrust::device_vector<uint32_t> feature_segments =
      std::vector<bst_row_t>{0, 2, 4};
  thrust::device_vector<float> feature_values =
      std::vector<float>{1.0, 2.0, 11.0, 12.0};
  thrust::device_vector<float> feature_min_values =
      std::vector<float>{0.0, 10.0};
  thrust::device_vector<GradientPairPrecise> feature_histogram =
      std::vector<GradientPairPrecise>{
          {-10.0, 0.5}, {10.0, 0.5}, {-0.5, 0.5}, {0.5, 0.5}};
  EvaluateSplitInputs input{1,0,
                                          parent_sum,
                                          dh::ToSpan(feature_set),
                                          dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{
      param,
                                          {},
                                          dh::ToSpan(feature_segments),
                                          dh::ToSpan(feature_values),
                                          dh::ToSpan(feature_min_values),
  };

  GPUHistEvaluator evaluator(tparam, feature_min_values.size(), 0);
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;

  EXPECT_EQ(result.findex, 1);
  EXPECT_EQ(result.fvalue, 11.0);
  EXPECT_EQ(result.left_sum, GradientPairPrecise(-0.5, 0.5));
  EXPECT_EQ(result.right_sum, GradientPairPrecise(0.5, 0.5));
}

// Features 0 and 1 have identical gain, the algorithm must select 0
TEST(GpuHist, EvaluateSingleSplitBreakTies) {
  GradientPairPrecise parent_sum(0.0, 1.0);
  TrainParam tparam = ZeroParam();
  tparam.UpdateAllowUnknown(Args{});
  GPUTrainingParam param{tparam};

  thrust::device_vector<bst_feature_t> feature_set =
      std::vector<bst_feature_t>{0, 1};
  thrust::device_vector<uint32_t> feature_segments =
      std::vector<bst_row_t>{0, 2, 4};
  thrust::device_vector<float> feature_values =
      std::vector<float>{1.0, 2.0, 11.0, 12.0};
  thrust::device_vector<float> feature_min_values =
      std::vector<float>{0.0, 10.0};
  thrust::device_vector<GradientPairPrecise> feature_histogram =
      std::vector<GradientPairPrecise>{
          {-0.5, 0.5}, {0.5, 0.5}, {-0.5, 0.5}, {0.5, 0.5}};
  EvaluateSplitInputs input{1,0,
                                          parent_sum,
                                          dh::ToSpan(feature_set),
                                          dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{
      param,
                                          {},
                                          dh::ToSpan(feature_segments),
                                          dh::ToSpan(feature_values),
                                          dh::ToSpan(feature_min_values),
  };

  GPUHistEvaluator evaluator(tparam, feature_min_values.size(), 0);
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input,shared_inputs).split;

  EXPECT_EQ(result.findex, 0);
  EXPECT_EQ(result.fvalue, 1.0);
}

TEST(GpuHist, EvaluateSplits) {
  thrust::device_vector<DeviceSplitCandidate> out_splits(2);
  GradientPairPrecise parent_sum(0.0, 1.0);
  TrainParam tparam = ZeroParam();
  tparam.UpdateAllowUnknown(Args{});
  GPUTrainingParam param{tparam};

  thrust::device_vector<bst_feature_t> feature_set =
      std::vector<bst_feature_t>{0, 1};
  thrust::device_vector<uint32_t> feature_segments =
      std::vector<bst_row_t>{0, 2, 4};
  thrust::device_vector<float> feature_values =
      std::vector<float>{1.0, 2.0, 11.0, 12.0};
  thrust::device_vector<float> feature_min_values =
      std::vector<float>{0.0, 0.0};
  thrust::device_vector<GradientPairPrecise> feature_histogram_left =
      std::vector<GradientPairPrecise>{
          {-0.5, 0.5}, {0.5, 0.5}, {-1.0, 0.5}, {1.0, 0.5}};
  thrust::device_vector<GradientPairPrecise> feature_histogram_right =
      std::vector<GradientPairPrecise>{
          {-1.0, 0.5}, {1.0, 0.5}, {-0.5, 0.5}, {0.5, 0.5}};
  EvaluateSplitInputs input_left{
      1,0,
      parent_sum,
      dh::ToSpan(feature_set),
      dh::ToSpan(feature_histogram_left)};
  EvaluateSplitInputs input_right{
      2,0,
      parent_sum,
      dh::ToSpan(feature_set),
      dh::ToSpan(feature_histogram_right)};
  EvaluateSplitSharedInputs shared_inputs{
      param,
                                          {},
                                          dh::ToSpan(feature_segments),
                                          dh::ToSpan(feature_values),
                                          dh::ToSpan(feature_min_values),
  };

  GPUHistEvaluator evaluator{
      tparam, static_cast<bst_feature_t>(feature_min_values.size()), 0};
  dh::device_vector<EvaluateSplitInputs> inputs = std::vector<EvaluateSplitInputs>{input_left,input_right};
  evaluator.LaunchEvaluateSplits(input_left.feature_set.size(),dh::ToSpan(inputs),shared_inputs, evaluator.GetEvaluator(),
                           dh::ToSpan(out_splits));

  DeviceSplitCandidate result_left = out_splits[0];
  EXPECT_EQ(result_left.findex, 1);
  EXPECT_EQ(result_left.fvalue, 11.0);

  DeviceSplitCandidate result_right = out_splits[1];
  EXPECT_EQ(result_right.findex, 0);
  EXPECT_EQ(result_right.fvalue, 1.0);
}

TEST_F(TestPartitionBasedSplit, GpuHist) {
  dh::device_vector<FeatureType> ft{std::vector<FeatureType>{FeatureType::kCategorical}};
  GPUHistEvaluator evaluator{param_, static_cast<bst_feature_t>(info_.num_col_), 0};

  cuts_.cut_ptrs_.SetDevice(0);
  cuts_.cut_values_.SetDevice(0);
  cuts_.min_vals_.SetDevice(0);

  evaluator.Reset(cuts_, dh::ToSpan(ft), info_.num_col_, param_, 0);

  dh::device_vector<GradientPairPrecise> d_hist(hist_[0].size());
  auto node_hist = hist_[0];
  dh::safe_cuda(hipMemcpy(d_hist.data().get(), node_hist.data(), node_hist.size_bytes(),
                           hipMemcpyHostToDevice));
  dh::device_vector<bst_feature_t> feature_set{std::vector<bst_feature_t>{0}};

  EvaluateSplitInputs input{0, 0, total_gpair_, dh::ToSpan(feature_set), dh::ToSpan(d_hist)};
  EvaluateSplitSharedInputs shared_inputs{
      GPUTrainingParam{param_},          dh::ToSpan(ft),
      cuts_.cut_ptrs_.ConstDeviceSpan(), cuts_.cut_values_.ConstDeviceSpan(),
      cuts_.min_vals_.ConstDeviceSpan(),
  };
  auto split = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
  ASSERT_NEAR(split.loss_chg, best_score_, 1e-16);
}
}  // namespace tree
}  // namespace xgboost
