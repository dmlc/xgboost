#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2023, XGBoost contributors
 */
#include <gtest/gtest.h>
#include <thrust/host_vector.h>

#include "../../../../src/tree/gpu_hist/evaluate_splits.cuh"
#include "../../helpers.h"
#include "../../histogram_helpers.h"
#include "../test_evaluate_splits.h"  // TestPartitionBasedSplit

namespace xgboost::tree {
namespace {
auto ZeroParam() {
  auto args = Args{{"min_child_weight", "0"}, {"lambda", "0"}};
  TrainParam tparam;
  tparam.UpdateAllowUnknown(args);
  return tparam;
}
}  // anonymous namespace

inline GradientQuantiser DummyRoundingFactor(Context const* ctx) {
  thrust::device_vector<GradientPair> gpair(1);
  gpair[0] = {1000.f, 1000.f};  // Tests should not exceed sum of 1000
  return {ctx, dh::ToSpan(gpair), MetaInfo()};
}

thrust::device_vector<GradientPairInt64> ConvertToInteger(Context const* ctx,
                                                          std::vector<GradientPairPrecise> x) {
  auto r = DummyRoundingFactor(ctx);
  std::vector<GradientPairInt64> y(x.size());
  for (std::size_t i = 0; i < x.size(); i++) {
    y[i] = r.ToFixedPoint(GradientPair(x[i]));
  }
  return y;
}

TEST_F(TestCategoricalSplitWithMissing, GPUHistEvaluator) {
  auto ctx = MakeCUDACtx(0);
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0};
  GPUTrainingParam param{param_};
  cuts_.cut_ptrs_.SetDevice(ctx.Device());
  cuts_.cut_values_.SetDevice(ctx.Device());
  cuts_.min_vals_.SetDevice(ctx.Device());
  thrust::device_vector<GradientPairInt64> feature_histogram{
      ConvertToInteger(&ctx, feature_histogram_)};

  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  auto d_feature_types = dh::ToSpan(feature_types);
  auto quantiser = DummyRoundingFactor(&ctx);
  EvaluateSplitInputs input{1, 0, quantiser.ToFixedPoint(parent_sum_), dh::ToSpan(feature_set),
                            dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          d_feature_types,
                                          cuts_.cut_ptrs_.ConstDeviceSpan(),
                                          cuts_.cut_values_.ConstDeviceSpan(),
                                          cuts_.min_vals_.ConstDeviceSpan(),
                                          false};

  GPUHistEvaluator evaluator{param_, static_cast<bst_feature_t>(feature_set.size()), ctx.Device()};

  evaluator.Reset(cuts_, dh::ToSpan(feature_types), feature_set.size(), param_, false,
                  ctx.Device());
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;

  ASSERT_EQ(result.thresh, 1);
  this->CheckResult(result.loss_chg, result.findex, result.fvalue, result.is_cat,
                    result.dir == kLeftDir, quantiser.ToFloatingPoint(result.left_sum),
                    quantiser.ToFloatingPoint(result.right_sum));
}

TEST(GpuHist, PartitionBasic) {
  auto ctx = MakeCUDACtx(0);
  TrainParam tparam = ZeroParam();
  tparam.max_cat_to_onehot = 0;
  GPUTrainingParam param{tparam};

  common::HistogramCuts cuts;
  cuts.cut_values_.HostVector() = std::vector<float>{0.0, 1.0, 2.0};
  cuts.cut_ptrs_.HostVector() = std::vector<uint32_t>{0, 3};
  cuts.min_vals_.HostVector() = std::vector<float>{0.0};
  cuts.cut_ptrs_.SetDevice(ctx.Device());
  cuts.cut_values_.SetDevice(ctx.Device());
  cuts.min_vals_.SetDevice(ctx.Device());
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0};

  thrust::device_vector<int> monotonic_constraints(feature_set.size(), 0);
  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  common::Span<FeatureType> d_feature_types;
  auto max_cat =
      *std::max_element(cuts.cut_values_.HostVector().begin(), cuts.cut_values_.HostVector().end());
  cuts.SetCategorical(true, max_cat);
  d_feature_types = dh::ToSpan(feature_types);
  auto quantiser = DummyRoundingFactor(&ctx);
  EvaluateSplitSharedInputs shared_inputs{
      param,
      quantiser,
      d_feature_types,
      cuts.cut_ptrs_.ConstDeviceSpan(),
      cuts.cut_values_.ConstDeviceSpan(),
      cuts.min_vals_.ConstDeviceSpan(),
      false,
  };

  GPUHistEvaluator evaluator{tparam, static_cast<bst_feature_t>(feature_set.size()), ctx.Device()};
  evaluator.Reset(cuts, dh::ToSpan(feature_types), feature_set.size(), tparam, false, ctx.Device());

  {
    // -1.0s go right
    // -3.0s go left
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-5.0, 3.0});
    auto feature_histogram = ConvertToInteger(&ctx, {{-1.0, 1.0}, {-1.0, 1.0}, {-3.0, 1.0}});
    EvaluateSplitInputs input{0, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_EQ(cats, std::bitset<32>("11000000000000000000000000000000"));
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }

  {
    // -1.0s go right
    // -3.0s go left
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-7.0, 3.0});
    auto feature_histogram = ConvertToInteger(&ctx, {{-1.0, 1.0}, {-3.0, 1.0}, {-3.0, 1.0}});
    EvaluateSplitInputs input{1, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_EQ(cats, std::bitset<32>("10000000000000000000000000000000"));
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }
  {
    // All -1.0, gain from splitting should be 0.0
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-3.0, 3.0});
    auto feature_histogram = ConvertToInteger(&ctx, {{-1.0, 1.0}, {-1.0, 1.0}, {-1.0, 1.0}});
    EvaluateSplitInputs input{2, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_FLOAT_EQ(result.loss_chg, 0.0f);
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }
  // With 3.0/3.0 missing values
  // Forward, first 2 categories are selected, while the last one go to left along with missing
  // value
  {
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{0.0, 6.0});
    auto feature_histogram = ConvertToInteger(&ctx, {{-1.0, 1.0}, {-1.0, 1.0}, {-1.0, 1.0}});
    EvaluateSplitInputs input{3, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(cats, std::bitset<32>("11000000000000000000000000000000"));
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }
  {
    // -1.0s go right
    // -3.0s go left
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-5.0, 3.0});
    auto feature_histogram = ConvertToInteger(&ctx, {{-1.0, 1.0}, {-3.0, 1.0}, {-1.0, 1.0}});
    EvaluateSplitInputs input{4, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_EQ(cats, std::bitset<32>("10100000000000000000000000000000"));
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }
  {
    // -1.0s go right
    // -3.0s go left
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-5.0, 3.0});
    auto feature_histogram = ConvertToInteger(&ctx, {{-3.0, 1.0}, {-1.0, 1.0}, {-3.0, 1.0}});
    EvaluateSplitInputs input{5, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(cats, std::bitset<32>("01000000000000000000000000000000"));
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }
}

TEST(GpuHist, PartitionTwoFeatures) {
  auto ctx = MakeCUDACtx(0);
  TrainParam tparam = ZeroParam();
  tparam.max_cat_to_onehot = 0;
  GPUTrainingParam param{tparam};

  common::HistogramCuts cuts;
  cuts.cut_values_.HostVector() = std::vector<float>{0.0, 1.0, 2.0, 0.0, 1.0, 2.0};
  cuts.cut_ptrs_.HostVector() = std::vector<uint32_t>{0, 3, 6};
  cuts.min_vals_.HostVector() = std::vector<float>{0.0, 0.0};
  cuts.cut_ptrs_.SetDevice(ctx.Device());
  cuts.cut_values_.SetDevice(ctx.Device());
  cuts.min_vals_.SetDevice(ctx.Device());
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0, 1};

  thrust::device_vector<int> monotonic_constraints(feature_set.size(), 0);
  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  common::Span<FeatureType> d_feature_types(dh::ToSpan(feature_types));
  auto max_cat =
      *std::max_element(cuts.cut_values_.HostVector().begin(), cuts.cut_values_.HostVector().end());
  cuts.SetCategorical(true, max_cat);

  auto quantiser = DummyRoundingFactor(&ctx);
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          d_feature_types,
                                          cuts.cut_ptrs_.ConstDeviceSpan(),
                                          cuts.cut_values_.ConstDeviceSpan(),
                                          cuts.min_vals_.ConstDeviceSpan(),
                                          false};

  GPUHistEvaluator evaluator{tparam, static_cast<bst_feature_t>(feature_set.size()), ctx.Device()};
  evaluator.Reset(cuts, dh::ToSpan(feature_types), feature_set.size(), tparam, false, ctx.Device());

  {
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-6.0, 3.0});
    auto feature_histogram = ConvertToInteger(
        &ctx, {{-2.0, 1.0}, {-2.0, 1.0}, {-2.0, 1.0}, {-1.0, 1.0}, {-1.0, 1.0}, {-4.0, 1.0}});
    EvaluateSplitInputs input{0, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.findex, 1);
    EXPECT_EQ(cats, std::bitset<32>("11000000000000000000000000000000"));
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }

  {
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-6.0, 3.0});
    auto feature_histogram = ConvertToInteger(
        &ctx, {{-2.0, 1.0}, {-2.0, 1.0}, {-2.0, 1.0}, {-1.0, 1.0}, {-2.5, 1.0}, {-2.5, 1.0}});
    EvaluateSplitInputs input{1, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.findex, 1);
    EXPECT_EQ(cats, std::bitset<32>("10000000000000000000000000000000"));
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }
}

TEST(GpuHist, PartitionTwoNodes) {
  auto ctx = MakeCUDACtx(0);
  TrainParam tparam = ZeroParam();
  tparam.max_cat_to_onehot = 0;
  GPUTrainingParam param{tparam};

  common::HistogramCuts cuts;
  cuts.cut_values_.HostVector() = std::vector<float>{0.0, 1.0, 2.0};
  cuts.cut_ptrs_.HostVector() = std::vector<uint32_t>{0, 3};
  cuts.min_vals_.HostVector() = std::vector<float>{0.0};
  cuts.cut_ptrs_.SetDevice(ctx.Device());
  cuts.cut_values_.SetDevice(ctx.Device());
  cuts.min_vals_.SetDevice(ctx.Device());
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0};

  thrust::device_vector<int> monotonic_constraints(feature_set.size(), 0);
  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  common::Span<FeatureType> d_feature_types(dh::ToSpan(feature_types));
  auto max_cat =
      *std::max_element(cuts.cut_values_.HostVector().begin(), cuts.cut_values_.HostVector().end());
  cuts.SetCategorical(true, max_cat);

  auto quantiser = DummyRoundingFactor(&ctx);
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          d_feature_types,
                                          cuts.cut_ptrs_.ConstDeviceSpan(),
                                          cuts.cut_values_.ConstDeviceSpan(),
                                          cuts.min_vals_.ConstDeviceSpan(),
                                          false};

  GPUHistEvaluator evaluator{tparam, static_cast<bst_feature_t>(feature_set.size()),
                             ctx.Device()};
  evaluator.Reset(cuts, dh::ToSpan(feature_types), feature_set.size(), tparam, false,
                  ctx.Device());

  {
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-6.0, 3.0});
    auto feature_histogram_a = ConvertToInteger(
        &ctx, {{-1.0, 1.0}, {-2.5, 1.0}, {-2.5, 1.0}, {-1.0, 1.0}, {-1.0, 1.0}, {-4.0, 1.0}});
    thrust::device_vector<EvaluateSplitInputs> inputs(2);
    inputs[0] = EvaluateSplitInputs{0, 0, parent_sum, dh::ToSpan(feature_set),
                                    dh::ToSpan(feature_histogram_a)};
    auto feature_histogram_b = ConvertToInteger(&ctx, {{-1.0, 1.0}, {-1.0, 1.0}, {-4.0, 1.0}});
    inputs[1] = EvaluateSplitInputs{1, 0, parent_sum, dh::ToSpan(feature_set),
                                    dh::ToSpan(feature_histogram_b)};
    thrust::device_vector<GPUExpandEntry> results(2);
    evaluator.EvaluateSplits(&ctx, {0, 1}, 1, dh::ToSpan(inputs), shared_inputs,
                             dh::ToSpan(results));
    EXPECT_EQ(std::bitset<32>(evaluator.GetHostNodeCats(0)[0]),
              std::bitset<32>("10000000000000000000000000000000"));
    EXPECT_EQ(std::bitset<32>(evaluator.GetHostNodeCats(1)[0]),
              std::bitset<32>("11000000000000000000000000000000"));
  }
}

void TestEvaluateSingleSplit(bool is_categorical) {
  auto ctx = MakeCUDACtx(0);
  auto quantiser = DummyRoundingFactor(&ctx);
  auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{0.0, 1.0});
  TrainParam tparam = ZeroParam();
  GPUTrainingParam param{tparam};

  common::HistogramCuts cuts{
      MakeCutsForTest({1.0, 2.0, 11.0, 12.0}, {0, 2, 4}, {0.0, 0.0}, ctx.Device())};
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0, 1};

  // Setup gradients so that second feature gets higher gain
  auto feature_histogram =
      ConvertToInteger(&ctx, {{-0.5, 0.5}, {0.5, 0.5}, {-1.0, 0.5}, {1.0, 0.5}});

  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  common::Span<FeatureType> d_feature_types;
  if (is_categorical) {
    auto max_cat = *std::max_element(cuts.cut_values_.HostVector().begin(),
                                     cuts.cut_values_.HostVector().end());
    cuts.SetCategorical(true, max_cat);
    d_feature_types = dh::ToSpan(feature_types);
  }

  EvaluateSplitInputs input{1, 0, parent_sum, dh::ToSpan(feature_set),
                            dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          d_feature_types,
                                          cuts.cut_ptrs_.ConstDeviceSpan(),
                                          cuts.cut_values_.ConstDeviceSpan(),
                                          cuts.min_vals_.ConstDeviceSpan(),
                                          false};

  GPUHistEvaluator evaluator{tparam, static_cast<bst_feature_t>(feature_set.size()),
                             ctx.Device()};
  evaluator.Reset(cuts, dh::ToSpan(feature_types), feature_set.size(), tparam, false,
                  ctx.Device());
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;

  EXPECT_EQ(result.findex, 1);
  if (is_categorical) {
    ASSERT_TRUE(std::isnan(result.fvalue));
  } else {
    EXPECT_EQ(result.fvalue, 11.0);
  }
  EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
}

TEST(GpuHist, EvaluateSingleSplit) { TestEvaluateSingleSplit(false); }

TEST(GpuHist, EvaluateSingleCategoricalSplit) { TestEvaluateSingleSplit(true); }

TEST(GpuHist, EvaluateSingleSplitMissing) {
  auto ctx = MakeCUDACtx(0);
  auto quantiser = DummyRoundingFactor(&ctx);
  auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{1.0, 1.5});
  TrainParam tparam = ZeroParam();
  GPUTrainingParam param{tparam};

  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0};
  thrust::device_vector<uint32_t> feature_segments = std::vector<bst_row_t>{0, 2};
  thrust::device_vector<float> feature_values = std::vector<float>{1.0, 2.0};
  thrust::device_vector<float> feature_min_values = std::vector<float>{0.0};
  auto feature_histogram = ConvertToInteger(&ctx, {{-0.5, 0.5}, {0.5, 0.5}});
  EvaluateSplitInputs input{1, 0, parent_sum, dh::ToSpan(feature_set),
                            dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          {},
                                          dh::ToSpan(feature_segments),
                                          dh::ToSpan(feature_values),
                                          dh::ToSpan(feature_min_values),
                                          false};

  GPUHistEvaluator evaluator(tparam, feature_set.size(), FstCU());
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;

  EXPECT_EQ(result.findex, 0);
  EXPECT_EQ(result.fvalue, 1.0);
  EXPECT_EQ(result.dir, kRightDir);
  EXPECT_EQ(result.left_sum, quantiser.ToFixedPoint(GradientPairPrecise(-0.5, 0.5)));
  EXPECT_EQ(result.right_sum, quantiser.ToFixedPoint(GradientPairPrecise(1.5, 1.0)));
}

TEST(GpuHist, EvaluateSingleSplitEmpty) {
  auto ctx = MakeCUDACtx(0);
  TrainParam tparam = ZeroParam();
  GPUHistEvaluator evaluator(tparam, 1, FstCU());
  DeviceSplitCandidate result =
      evaluator
          .EvaluateSingleSplit(
              &ctx, EvaluateSplitInputs{},
              EvaluateSplitSharedInputs{
                  GPUTrainingParam(tparam), DummyRoundingFactor(&ctx), {}, {}, {}, {}, false})
          .split;
  EXPECT_EQ(result.findex, -1);
  EXPECT_LT(result.loss_chg, 0.0f);
}

// Feature 0 has a better split, but the algorithm must select feature 1
TEST(GpuHist, EvaluateSingleSplitFeatureSampling) {
  auto ctx = MakeCUDACtx(0);
  auto quantiser = DummyRoundingFactor(&ctx);
  auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{0.0, 1.0});
  TrainParam tparam = ZeroParam();
  tparam.UpdateAllowUnknown(Args{});
  GPUTrainingParam param{tparam};

  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{1};
  thrust::device_vector<uint32_t> feature_segments = std::vector<bst_row_t>{0, 2, 4};
  thrust::device_vector<float> feature_values = std::vector<float>{1.0, 2.0, 11.0, 12.0};
  thrust::device_vector<float> feature_min_values = std::vector<float>{0.0, 10.0};
  auto feature_histogram =
      ConvertToInteger(&ctx, {{-10.0, 0.5}, {10.0, 0.5}, {-0.5, 0.5}, {0.5, 0.5}});
  EvaluateSplitInputs input{1, 0, parent_sum, dh::ToSpan(feature_set),
                            dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          {},
                                          dh::ToSpan(feature_segments),
                                          dh::ToSpan(feature_values),
                                          dh::ToSpan(feature_min_values),
                                          false};

  GPUHistEvaluator evaluator(tparam, feature_min_values.size(), FstCU());
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;

  EXPECT_EQ(result.findex, 1);
  EXPECT_EQ(result.fvalue, 11.0);
  EXPECT_EQ(result.left_sum, quantiser.ToFixedPoint(GradientPairPrecise(-0.5, 0.5)));
  EXPECT_EQ(result.right_sum, quantiser.ToFixedPoint(GradientPairPrecise(0.5, 0.5)));
}

// Features 0 and 1 have identical gain, the algorithm must select 0
TEST(GpuHist, EvaluateSingleSplitBreakTies) {
  auto ctx = MakeCUDACtx(0);
  auto quantiser = DummyRoundingFactor(&ctx);
  auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{0.0, 1.0});
  TrainParam tparam = ZeroParam();
  tparam.UpdateAllowUnknown(Args{});
  GPUTrainingParam param{tparam};

  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0, 1};
  thrust::device_vector<uint32_t> feature_segments = std::vector<bst_row_t>{0, 2, 4};
  thrust::device_vector<float> feature_values = std::vector<float>{1.0, 2.0, 11.0, 12.0};
  thrust::device_vector<float> feature_min_values = std::vector<float>{0.0, 10.0};
  auto feature_histogram =
      ConvertToInteger(&ctx, {{-0.5, 0.5}, {0.5, 0.5}, {-0.5, 0.5}, {0.5, 0.5}});
  EvaluateSplitInputs input{1, 0, parent_sum, dh::ToSpan(feature_set),
                            dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          {},
                                          dh::ToSpan(feature_segments),
                                          dh::ToSpan(feature_values),
                                          dh::ToSpan(feature_min_values),
                                          false};

  GPUHistEvaluator evaluator(tparam, feature_min_values.size(), FstCU());
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;

  EXPECT_EQ(result.findex, 0);
  EXPECT_EQ(result.fvalue, 1.0);
}

TEST(GpuHist, EvaluateSplits) {
  auto ctx = MakeCUDACtx(0);
  thrust::device_vector<DeviceSplitCandidate> out_splits(2);
  auto quantiser = DummyRoundingFactor(&ctx);
  auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{0.0, 1.0});
  TrainParam tparam = ZeroParam();
  tparam.UpdateAllowUnknown(Args{});
  GPUTrainingParam param{tparam};

  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0, 1};
  thrust::device_vector<uint32_t> feature_segments = std::vector<bst_row_t>{0, 2, 4};
  thrust::device_vector<float> feature_values = std::vector<float>{1.0, 2.0, 11.0, 12.0};
  thrust::device_vector<float> feature_min_values = std::vector<float>{0.0, 0.0};
  auto feature_histogram_left =
      ConvertToInteger(&ctx, {{-0.5, 0.5}, {0.5, 0.5}, {-1.0, 0.5}, {1.0, 0.5}});
  auto feature_histogram_right =
      ConvertToInteger(&ctx, {{-1.0, 0.5}, {1.0, 0.5}, {-0.5, 0.5}, {0.5, 0.5}});
  EvaluateSplitInputs input_left{1, 0, parent_sum, dh::ToSpan(feature_set),
                                 dh::ToSpan(feature_histogram_left)};
  EvaluateSplitInputs input_right{2, 0, parent_sum, dh::ToSpan(feature_set),
                                  dh::ToSpan(feature_histogram_right)};
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          {},
                                          dh::ToSpan(feature_segments),
                                          dh::ToSpan(feature_values),
                                          dh::ToSpan(feature_min_values),
                                          false};

  GPUHistEvaluator evaluator{tparam, static_cast<bst_feature_t>(feature_min_values.size()),
                             FstCU()};
  dh::device_vector<EvaluateSplitInputs> inputs =
      std::vector<EvaluateSplitInputs>{input_left, input_right};
  evaluator.LaunchEvaluateSplits(input_left.feature_set.size(), dh::ToSpan(inputs), shared_inputs,
                                 evaluator.GetEvaluator(), dh::ToSpan(out_splits));

  DeviceSplitCandidate result_left = out_splits[0];
  EXPECT_EQ(result_left.findex, 1);
  EXPECT_EQ(result_left.fvalue, 11.0);

  DeviceSplitCandidate result_right = out_splits[1];
  EXPECT_EQ(result_right.findex, 0);
  EXPECT_EQ(result_right.fvalue, 1.0);
}

TEST_F(TestPartitionBasedSplit, GpuHist) {
  auto ctx = MakeCUDACtx(0);
  dh::device_vector<FeatureType> ft{std::vector<FeatureType>{FeatureType::kCategorical}};
  GPUHistEvaluator evaluator{param_, static_cast<bst_feature_t>(info_.num_col_), ctx.Device()};

  cuts_.cut_ptrs_.SetDevice(ctx.Device());
  cuts_.cut_values_.SetDevice(ctx.Device());
  cuts_.min_vals_.SetDevice(ctx.Device());

  evaluator.Reset(cuts_, dh::ToSpan(ft), info_.num_col_, param_, false, ctx.Device());

  // Convert the sample histogram to fixed point
  auto quantiser = DummyRoundingFactor(&ctx);
  thrust::host_vector<GradientPairInt64> h_hist;
  for (auto e : hist_[0]) {
    h_hist.push_back(quantiser.ToFixedPoint(e));
  }
  dh::device_vector<GradientPairInt64> d_hist = h_hist;
  dh::device_vector<bst_feature_t> feature_set{std::vector<bst_feature_t>{0}};

  EvaluateSplitInputs input{0, 0, quantiser.ToFixedPoint(total_gpair_), dh::ToSpan(feature_set),
                            dh::ToSpan(d_hist)};
  EvaluateSplitSharedInputs shared_inputs{GPUTrainingParam{param_},
                                          quantiser,
                                          dh::ToSpan(ft),
                                          cuts_.cut_ptrs_.ConstDeviceSpan(),
                                          cuts_.cut_values_.ConstDeviceSpan(),
                                          cuts_.min_vals_.ConstDeviceSpan(),
                                          false};
  auto split = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;
  ASSERT_NEAR(split.loss_chg, best_score_, 1e-2);
}

class MGPUHistTest : public BaseMGPUTest {};

namespace {
void VerifyColumnSplitEvaluateSingleSplit(bool is_categorical) {
  auto ctx = MakeCUDACtx(GPUIDX);
  auto rank = collective::GetRank();
  auto quantiser = DummyRoundingFactor(&ctx);
  auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{0.0, 1.0});
  TrainParam tparam = ZeroParam();
  GPUTrainingParam param{tparam};

  common::HistogramCuts cuts{
      rank == 0 ? MakeCutsForTest({1.0, 2.0}, {0, 2, 2}, {0.0, 0.0}, ctx.Device())
                : MakeCutsForTest({11.0, 12.0}, {0, 0, 2}, {0.0, 0.0}, ctx.Device())};
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0, 1};

  // Setup gradients so that second feature gets higher gain
  auto feature_histogram = rank == 0 ? ConvertToInteger(&ctx, {{-0.5, 0.5}, {0.5, 0.5}})
                                     : ConvertToInteger(&ctx, {{-1.0, 0.5}, {1.0, 0.5}});

  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  common::Span<FeatureType> d_feature_types;
  if (is_categorical) {
    auto max_cat = *std::max_element(cuts.cut_values_.HostVector().begin(),
                                     cuts.cut_values_.HostVector().end());
    cuts.SetCategorical(true, max_cat);
    d_feature_types = dh::ToSpan(feature_types);
  }

  EvaluateSplitInputs input{1, 0, parent_sum, dh::ToSpan(feature_set),
                            dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          d_feature_types,
                                          cuts.cut_ptrs_.ConstDeviceSpan(),
                                          cuts.cut_values_.ConstDeviceSpan(),
                                          cuts.min_vals_.ConstDeviceSpan(),
                                          false};

  GPUHistEvaluator evaluator{tparam, static_cast<bst_feature_t>(feature_set.size()), ctx.Device()};
  evaluator.Reset(cuts, dh::ToSpan(feature_types), feature_set.size(), tparam, true, ctx.Device());
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(&ctx, input, shared_inputs).split;

  EXPECT_EQ(result.findex, 1) << "rank: " << rank;
  if (is_categorical) {
    ASSERT_TRUE(std::isnan(result.fvalue));
  } else {
    EXPECT_EQ(result.fvalue, 11.0) << "rank: " << rank;
  }
  EXPECT_EQ(result.left_sum + result.right_sum, parent_sum) << "rank: " << rank;
}
}  // anonymous namespace

TEST_F(MGPUHistTest, ColumnSplitEvaluateSingleSplit) {
  DoTest(VerifyColumnSplitEvaluateSingleSplit, false);
}

TEST_F(MGPUHistTest, ColumnSplitEvaluateSingleCategoricalSplit) {
  DoTest(VerifyColumnSplitEvaluateSingleSplit, true);
}
}  // namespace xgboost::tree
