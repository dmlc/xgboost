#include "hip/hip_runtime.h"
/*!
 * Copyright 2020-2022 by XGBoost contributors
 */
#include <gtest/gtest.h>

#include "../../../../src/tree/gpu_hist/evaluate_splits.cuh"
#include "../../helpers.h"
#include "../../histogram_helpers.h"
#include "../test_evaluate_splits.h"  // TestPartitionBasedSplit
#include <thrust/host_vector.h>

namespace xgboost {
namespace tree {
namespace {
auto ZeroParam() {
  auto args = Args{{"min_child_weight", "0"},
                   {"lambda", "0"}};
  TrainParam tparam;
  tparam.UpdateAllowUnknown(args);
  return tparam;
}

}  // anonymous namespace

inline GradientQuantiser DummyRoundingFactor() {
  thrust::device_vector<GradientPair> gpair(1);
  gpair[0] = {1000.f, 1000.f};  // Tests should not exceed sum of 1000
  return GradientQuantiser(dh::ToSpan(gpair));
}

thrust::device_vector<GradientPairInt64> ConvertToInteger(std::vector<GradientPairPrecise> x) {
  auto r = DummyRoundingFactor();
  std::vector<GradientPairInt64> y(x.size());
  for (std::size_t i = 0; i < x.size(); i++) {
    y[i] = r.ToFixedPoint(GradientPair(x[i]));
  }
  return y;
}


TEST_F(TestCategoricalSplitWithMissing, GPUHistEvaluator) {
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0};
  GPUTrainingParam param{param_};
  cuts_.cut_ptrs_.SetDevice(0);
  cuts_.cut_values_.SetDevice(0);
  cuts_.min_vals_.SetDevice(0);
  thrust::device_vector<GradientPairInt64> feature_histogram{ConvertToInteger(feature_histogram_)};

  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  auto d_feature_types = dh::ToSpan(feature_types);
  auto quantiser = DummyRoundingFactor();
  EvaluateSplitInputs input{1, 0, quantiser.ToFixedPoint(parent_sum_), dh::ToSpan(feature_set),
                            dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          d_feature_types,
                                          cuts_.cut_ptrs_.ConstDeviceSpan(),
                                          cuts_.cut_values_.ConstDeviceSpan(),
                                          cuts_.min_vals_.ConstDeviceSpan(),
                                          false};

  GPUHistEvaluator evaluator{param_, static_cast<bst_feature_t>(feature_set.size()), 0};

  evaluator.Reset(cuts_, dh::ToSpan(feature_types), feature_set.size(), param_, 0);
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;

  ASSERT_EQ(result.thresh, 1);
  this->CheckResult(result.loss_chg, result.findex, result.fvalue, result.is_cat,
                    result.dir == kLeftDir, quantiser.ToFloatingPoint(result.left_sum), quantiser.ToFloatingPoint(result.right_sum));
}

TEST(GpuHist, PartitionBasic) {
  TrainParam tparam = ZeroParam();
  tparam.max_cat_to_onehot = 0;
  GPUTrainingParam param{tparam};

  common::HistogramCuts cuts;
  cuts.cut_values_.HostVector() = std::vector<float>{0.0, 1.0, 2.0};
  cuts.cut_ptrs_.HostVector() = std::vector<uint32_t>{0, 3};
  cuts.min_vals_.HostVector() = std::vector<float>{0.0};
  cuts.cut_ptrs_.SetDevice(0);
  cuts.cut_values_.SetDevice(0);
  cuts.min_vals_.SetDevice(0);
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0};

  thrust::device_vector<int> monotonic_constraints(feature_set.size(), 0);
  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  common::Span<FeatureType> d_feature_types;
  auto max_cat =
      *std::max_element(cuts.cut_values_.HostVector().begin(), cuts.cut_values_.HostVector().end());
  cuts.SetCategorical(true, max_cat);
  d_feature_types = dh::ToSpan(feature_types);
  auto quantiser = DummyRoundingFactor();
  EvaluateSplitSharedInputs shared_inputs{
      param,
      quantiser,
      d_feature_types,
      cuts.cut_ptrs_.ConstDeviceSpan(),
      cuts.cut_values_.ConstDeviceSpan(),
      cuts.min_vals_.ConstDeviceSpan(),
      false,
  };

  GPUHistEvaluator evaluator{tparam, static_cast<bst_feature_t>(feature_set.size()), 0};
  evaluator.Reset(cuts, dh::ToSpan(feature_types), feature_set.size(), tparam, 0);

  {
    // -1.0s go right
    // -3.0s go left
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-5.0, 3.0});
    auto feature_histogram = ConvertToInteger({{-1.0, 1.0}, {-1.0, 1.0}, {-3.0, 1.0}});
    EvaluateSplitInputs input{0, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_EQ(cats, std::bitset<32>("11000000000000000000000000000000"));
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }

  {
    // -1.0s go right
    // -3.0s go left
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-7.0, 3.0});
    auto feature_histogram = ConvertToInteger({{-1.0, 1.0}, {-3.0, 1.0}, {-3.0, 1.0}});
    EvaluateSplitInputs input{1, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_EQ(cats, std::bitset<32>("10000000000000000000000000000000"));
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }
  {
    // All -1.0, gain from splitting should be 0.0
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-3.0, 3.0});
    auto feature_histogram = ConvertToInteger({{-1.0, 1.0}, {-1.0, 1.0}, {-1.0, 1.0}});
    EvaluateSplitInputs input{2, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_FLOAT_EQ(result.loss_chg, 0.0f);
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }
  // With 3.0/3.0 missing values
  // Forward, first 2 categories are selected, while the last one go to left along with missing value
  {
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{0.0, 6.0});
    auto feature_histogram = ConvertToInteger({{-1.0, 1.0}, {-1.0, 1.0}, {-1.0, 1.0}});
    EvaluateSplitInputs input{3, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(cats, std::bitset<32>("11000000000000000000000000000000"));
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }
  {
    // -1.0s go right
    // -3.0s go left
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-5.0, 3.0});
    auto feature_histogram = ConvertToInteger({{-1.0, 1.0}, {-3.0, 1.0}, {-1.0, 1.0}});
    EvaluateSplitInputs input{4, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.dir, kLeftDir);
    EXPECT_EQ(cats, std::bitset<32>("10100000000000000000000000000000"));
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }
  {
    // -1.0s go right
    // -3.0s go left
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-5.0, 3.0});
    auto feature_histogram = ConvertToInteger({{-3.0, 1.0}, {-1.0, 1.0}, {-3.0, 1.0}});
    EvaluateSplitInputs input{5, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(cats, std::bitset<32>("01000000000000000000000000000000"));
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }
}

TEST(GpuHist, PartitionTwoFeatures) {
  TrainParam tparam = ZeroParam();
  tparam.max_cat_to_onehot = 0;
  GPUTrainingParam param{tparam};

  common::HistogramCuts cuts;
  cuts.cut_values_.HostVector() = std::vector<float>{0.0, 1.0, 2.0, 0.0, 1.0, 2.0};
  cuts.cut_ptrs_.HostVector() = std::vector<uint32_t>{0, 3, 6};
  cuts.min_vals_.HostVector() = std::vector<float>{0.0, 0.0};
  cuts.cut_ptrs_.SetDevice(0);
  cuts.cut_values_.SetDevice(0);
  cuts.min_vals_.SetDevice(0);
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0, 1};

  thrust::device_vector<int> monotonic_constraints(feature_set.size(), 0);
  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  common::Span<FeatureType> d_feature_types(dh::ToSpan(feature_types));
  auto max_cat =
      *std::max_element(cuts.cut_values_.HostVector().begin(), cuts.cut_values_.HostVector().end());
  cuts.SetCategorical(true, max_cat);

  auto quantiser = DummyRoundingFactor();
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          d_feature_types,
                                          cuts.cut_ptrs_.ConstDeviceSpan(),
                                          cuts.cut_values_.ConstDeviceSpan(),
                                          cuts.min_vals_.ConstDeviceSpan(),
                                          false};

  GPUHistEvaluator evaluator{tparam, static_cast<bst_feature_t>(feature_set.size()), 0};
  evaluator.Reset(cuts, dh::ToSpan(feature_types), feature_set.size(), tparam, 0);

  {
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-6.0, 3.0});
    auto feature_histogram = ConvertToInteger({        {-2.0, 1.0}, {-2.0, 1.0}, {-2.0, 1.0}, {-1.0, 1.0}, {-1.0, 1.0}, {-4.0, 1.0}});
    EvaluateSplitInputs input{0, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.findex, 1);
    EXPECT_EQ(cats, std::bitset<32>("11000000000000000000000000000000"));
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }

  {
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-6.0, 3.0});
    auto feature_histogram = ConvertToInteger({        {-2.0, 1.0}, {-2.0, 1.0}, {-2.0, 1.0}, {-1.0, 1.0}, {-2.5, 1.0}, {-2.5, 1.0}});
    EvaluateSplitInputs input{1, 0, parent_sum, dh::ToSpan(feature_set),
                              dh::ToSpan(feature_histogram)};
    DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
    auto cats = std::bitset<32>(evaluator.GetHostNodeCats(input.nidx)[0]);
    EXPECT_EQ(result.findex, 1);
    EXPECT_EQ(cats, std::bitset<32>("10000000000000000000000000000000"));
    EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
  }
}

TEST(GpuHist, PartitionTwoNodes) {
  TrainParam tparam = ZeroParam();
  tparam.max_cat_to_onehot = 0;
  GPUTrainingParam param{tparam};

  common::HistogramCuts cuts;
  cuts.cut_values_.HostVector() = std::vector<float>{0.0, 1.0, 2.0};
  cuts.cut_ptrs_.HostVector() = std::vector<uint32_t>{0, 3};
  cuts.min_vals_.HostVector() = std::vector<float>{0.0};
  cuts.cut_ptrs_.SetDevice(0);
  cuts.cut_values_.SetDevice(0);
  cuts.min_vals_.SetDevice(0);
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0};

  thrust::device_vector<int> monotonic_constraints(feature_set.size(), 0);
  dh::device_vector<FeatureType> feature_types(feature_set.size(), FeatureType::kCategorical);
  common::Span<FeatureType> d_feature_types(dh::ToSpan(feature_types));
  auto max_cat =
      *std::max_element(cuts.cut_values_.HostVector().begin(), cuts.cut_values_.HostVector().end());
  cuts.SetCategorical(true, max_cat);

  auto quantiser = DummyRoundingFactor();
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          d_feature_types,
                                          cuts.cut_ptrs_.ConstDeviceSpan(),
                                          cuts.cut_values_.ConstDeviceSpan(),
                                          cuts.min_vals_.ConstDeviceSpan(),
                                          false};

  GPUHistEvaluator evaluator{tparam, static_cast<bst_feature_t>(feature_set.size()), 0};
  evaluator.Reset(cuts, dh::ToSpan(feature_types), feature_set.size(), tparam, 0);

  {
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{-6.0, 3.0});
    auto feature_histogram_a = ConvertToInteger({{-1.0, 1.0}, {-2.5, 1.0}, {-2.5, 1.0},
                                         {-1.0, 1.0}, {-1.0, 1.0}, {-4.0, 1.0}});
    thrust::device_vector<EvaluateSplitInputs> inputs(2);
    inputs[0] = EvaluateSplitInputs{0, 0, parent_sum, dh::ToSpan(feature_set),
                                    dh::ToSpan(feature_histogram_a)};
    auto feature_histogram_b = ConvertToInteger({{-1.0, 1.0}, {-1.0, 1.0}, {-4.0, 1.0}});
    inputs[1] = EvaluateSplitInputs{1, 0, parent_sum, dh::ToSpan(feature_set),
                                    dh::ToSpan(feature_histogram_b)};
    thrust::device_vector<GPUExpandEntry> results(2);
    evaluator.EvaluateSplits({0, 1}, 1, dh::ToSpan(inputs), shared_inputs, dh::ToSpan(results));
    GPUExpandEntry result_a = results[0];
    GPUExpandEntry result_b = results[1];
    EXPECT_EQ(std::bitset<32>(evaluator.GetHostNodeCats(0)[0]),
              std::bitset<32>("10000000000000000000000000000000"));
    EXPECT_EQ(std::bitset<32>(evaluator.GetHostNodeCats(1)[0]),
              std::bitset<32>("11000000000000000000000000000000"));
  }
}

void TestEvaluateSingleSplit(bool is_categorical) {
  auto quantiser = DummyRoundingFactor();
  auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{0.0, 1.0});
  TrainParam tparam = ZeroParam();
  GPUTrainingParam param{tparam};

  common::HistogramCuts cuts{MakeCutsForTest({1.0, 2.0, 11.0, 12.0}, {0, 2, 4}, {0.0, 0.0}, 0)};
  thrust::device_vector<bst_feature_t> feature_set = std::vector<bst_feature_t>{0, 1};

  // Setup gradients so that second feature gets higher gain
  auto feature_histogram = ConvertToInteger({          {-0.5, 0.5}, {0.5, 0.5}, {-1.0, 0.5}, {1.0, 0.5}});

  dh::device_vector<FeatureType> feature_types(feature_set.size(),
                                               FeatureType::kCategorical);
  common::Span<FeatureType> d_feature_types;
  if (is_categorical) {
    auto max_cat = *std::max_element(cuts.cut_values_.HostVector().begin(),
                                     cuts.cut_values_.HostVector().end());
    cuts.SetCategorical(true, max_cat);
    d_feature_types = dh::ToSpan(feature_types);
  }

  EvaluateSplitInputs input{1, 0, parent_sum, dh::ToSpan(feature_set),
                            dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          d_feature_types,
                                          cuts.cut_ptrs_.ConstDeviceSpan(),
                                          cuts.cut_values_.ConstDeviceSpan(),
                                          cuts.min_vals_.ConstDeviceSpan(),
                                          false};

  GPUHistEvaluator evaluator{
      tparam, static_cast<bst_feature_t>(feature_set.size()), 0};
  evaluator.Reset(cuts, dh::ToSpan(feature_types), feature_set.size(), tparam, 0);
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;

  EXPECT_EQ(result.findex, 1);
  if (is_categorical) {
    ASSERT_TRUE(std::isnan(result.fvalue));
  } else {
    EXPECT_EQ(result.fvalue, 11.0);
  }
  EXPECT_EQ(result.left_sum + result.right_sum, parent_sum);
}

TEST(GpuHist, EvaluateSingleSplit) {
  TestEvaluateSingleSplit(false);
}

TEST(GpuHist, EvaluateSingleCategoricalSplit) {
  TestEvaluateSingleSplit(true);
}

TEST(GpuHist, EvaluateSingleSplitMissing) {
  auto quantiser = DummyRoundingFactor();
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{1.0, 1.5});
  TrainParam tparam = ZeroParam();
  GPUTrainingParam param{tparam};

  thrust::device_vector<bst_feature_t> feature_set =
      std::vector<bst_feature_t>{0};
  thrust::device_vector<uint32_t> feature_segments =
      std::vector<bst_row_t>{0, 2};
  thrust::device_vector<float> feature_values = std::vector<float>{1.0, 2.0};
  thrust::device_vector<float> feature_min_values = std::vector<float>{0.0};
  auto feature_histogram = ConvertToInteger({{-0.5, 0.5}, {0.5, 0.5}});
  EvaluateSplitInputs input{1,0,
                                          parent_sum,
                                          dh::ToSpan(feature_set),
                                          dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          {},
                                          dh::ToSpan(feature_segments),
                                          dh::ToSpan(feature_values),
                                          dh::ToSpan(feature_min_values),
                                          false};

  GPUHistEvaluator evaluator(tparam, feature_set.size(), 0);
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;

  EXPECT_EQ(result.findex, 0);
  EXPECT_EQ(result.fvalue, 1.0);
  EXPECT_EQ(result.dir, kRightDir);
  EXPECT_EQ(result.left_sum,quantiser.ToFixedPoint(GradientPairPrecise(-0.5, 0.5)));
  EXPECT_EQ(result.right_sum, quantiser.ToFixedPoint(GradientPairPrecise(1.5, 1.0)));
}

TEST(GpuHist, EvaluateSingleSplitEmpty) {
  TrainParam tparam = ZeroParam();
  GPUHistEvaluator evaluator(tparam, 1, 0);
  DeviceSplitCandidate result =
      evaluator
          .EvaluateSingleSplit(
              EvaluateSplitInputs{},
              EvaluateSplitSharedInputs{
                  GPUTrainingParam(tparam), DummyRoundingFactor(), {}, {}, {}, {}, false})
          .split;
  EXPECT_EQ(result.findex, -1);
  EXPECT_LT(result.loss_chg, 0.0f);
}

// Feature 0 has a better split, but the algorithm must select feature 1
TEST(GpuHist, EvaluateSingleSplitFeatureSampling) {
  auto quantiser = DummyRoundingFactor();
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{0.0, 1.0});
  TrainParam tparam = ZeroParam();
  tparam.UpdateAllowUnknown(Args{});
  GPUTrainingParam param{tparam};

  thrust::device_vector<bst_feature_t> feature_set =
      std::vector<bst_feature_t>{1};
  thrust::device_vector<uint32_t> feature_segments =
      std::vector<bst_row_t>{0, 2, 4};
  thrust::device_vector<float> feature_values =
      std::vector<float>{1.0, 2.0, 11.0, 12.0};
  thrust::device_vector<float> feature_min_values =
      std::vector<float>{0.0, 10.0};
  auto feature_histogram = ConvertToInteger({          {-10.0, 0.5}, {10.0, 0.5}, {-0.5, 0.5}, {0.5, 0.5}});
  EvaluateSplitInputs input{1,0,
                                          parent_sum,
                                          dh::ToSpan(feature_set),
                                          dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          {},
                                          dh::ToSpan(feature_segments),
                                          dh::ToSpan(feature_values),
                                          dh::ToSpan(feature_min_values),
                                          false};

  GPUHistEvaluator evaluator(tparam, feature_min_values.size(), 0);
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input, shared_inputs).split;

  EXPECT_EQ(result.findex, 1);
  EXPECT_EQ(result.fvalue, 11.0);
  EXPECT_EQ(result.left_sum,quantiser.ToFixedPoint(GradientPairPrecise(-0.5, 0.5)));
  EXPECT_EQ(result.right_sum, quantiser.ToFixedPoint(GradientPairPrecise(0.5, 0.5)));
}

// Features 0 and 1 have identical gain, the algorithm must select 0
TEST(GpuHist, EvaluateSingleSplitBreakTies) {
  auto quantiser = DummyRoundingFactor();
    auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{0.0, 1.0});
  TrainParam tparam = ZeroParam();
  tparam.UpdateAllowUnknown(Args{});
  GPUTrainingParam param{tparam};

  thrust::device_vector<bst_feature_t> feature_set =
      std::vector<bst_feature_t>{0, 1};
  thrust::device_vector<uint32_t> feature_segments =
      std::vector<bst_row_t>{0, 2, 4};
  thrust::device_vector<float> feature_values =
      std::vector<float>{1.0, 2.0, 11.0, 12.0};
  thrust::device_vector<float> feature_min_values =
      std::vector<float>{0.0, 10.0};
  auto feature_histogram = ConvertToInteger({          {-0.5, 0.5}, {0.5, 0.5}, {-0.5, 0.5}, {0.5, 0.5}});
  EvaluateSplitInputs input{1,0,
                                          parent_sum,
                                          dh::ToSpan(feature_set),
                                          dh::ToSpan(feature_histogram)};
  EvaluateSplitSharedInputs shared_inputs{param,
                                          quantiser,
                                          {},
                                          dh::ToSpan(feature_segments),
                                          dh::ToSpan(feature_values),
                                          dh::ToSpan(feature_min_values),
                                          false};

  GPUHistEvaluator evaluator(tparam, feature_min_values.size(), 0);
  DeviceSplitCandidate result = evaluator.EvaluateSingleSplit(input,shared_inputs).split;

  EXPECT_EQ(result.findex, 0);
  EXPECT_EQ(result.fvalue, 1.0);
}

TEST(GpuHist, EvaluateSplits) {
  thrust::device_vector<DeviceSplitCandidate> out_splits(2);
  auto quantiser = DummyRoundingFactor();
  auto parent_sum = quantiser.ToFixedPoint(GradientPairPrecise{0.0, 1.0});
  TrainParam tparam = ZeroParam();
  tparam.UpdateAllowUnknown(Args{});
  GPUTrainingParam param{tparam};

  thrust::device_vector<bst_feature_t> feature_set =
      std::vector<bst_feature_t>{0, 1};
  thrust::device_vector<uint32_t> feature_segments =
      std::vector<bst_row_t>{0, 2, 4};
  thrust::device_vector<float> feature_values =
      std::vector<float>{1.0, 2.0, 11.0, 12.0};
  thrust::device_vector<float> feature_min_values =
      std::vector<float>{0.0, 0.0};
  auto feature_histogram_left = ConvertToInteger({          {-0.5, 0.5}, {0.5, 0.5}, {-1.0, 0.5}, {1.0, 0.5}});
  auto feature_histogram_right = ConvertToInteger({          {-1.0, 0.5}, {1.0, 0.5}, {-0.5, 0.5}, {0.5, 0.5}});
  EvaluateSplitInputs input_left{
      1,0,
      parent_sum,
      dh::ToSpan(feature_set),
      dh::ToSpan(feature_histogram_left)};
  EvaluateSplitInputs input_right{
      2,0,
      parent_sum,
      dh::ToSpan(feature_set),
      dh::ToSpan(feature_histogram_right)};
  EvaluateSplitSharedInputs shared_inputs{
      param,
      quantiser,
      {},
      dh::ToSpan(feature_segments),
      dh::ToSpan(feature_values),
      dh::ToSpan(feature_min_values),
      false
  };

  GPUHistEvaluator evaluator{
      tparam, static_cast<bst_feature_t>(feature_min_values.size()), 0};
  dh::device_vector<EvaluateSplitInputs> inputs = std::vector<EvaluateSplitInputs>{input_left,input_right};
  evaluator.LaunchEvaluateSplits(input_left.feature_set.size(),dh::ToSpan(inputs),shared_inputs, evaluator.GetEvaluator(),
                           dh::ToSpan(out_splits));

  DeviceSplitCandidate result_left = out_splits[0];
  EXPECT_EQ(result_left.findex, 1);
  EXPECT_EQ(result_left.fvalue, 11.0);

  DeviceSplitCandidate result_right = out_splits[1];
  EXPECT_EQ(result_right.findex, 0);
  EXPECT_EQ(result_right.fvalue, 1.0);
}

TEST_F(TestPartitionBasedSplit, GpuHist) {
  dh::device_vector<FeatureType> ft{std::vector<FeatureType>{FeatureType::kCategorical}};
  GPUHistEvaluator evaluator{param_, static_cast<bst_feature_t>(info_.num_col_), 0};

  cuts_.cut_ptrs_.SetDevice(0);
  cuts_.cut_values_.SetDevice(0);
  cuts_.min_vals_.SetDevice(0);

  evaluator.Reset(cuts_, dh::ToSpan(ft), info_.num_col_, param_, 0);

  // Convert the sample histogram to fixed point
  auto quantiser = DummyRoundingFactor();
  thrust::host_vector<GradientPairInt64> h_hist;
  for(auto e: hist_[0]){
    h_hist.push_back(quantiser.ToFixedPoint(e));
  }
  dh::device_vector<GradientPairInt64> d_hist = h_hist;
  dh::device_vector<bst_feature_t> feature_set{std::vector<bst_feature_t>{0}};

  EvaluateSplitInputs input{0, 0, quantiser.ToFixedPoint(total_gpair_), dh::ToSpan(feature_set), dh::ToSpan(d_hist)};
  EvaluateSplitSharedInputs shared_inputs{GPUTrainingParam{param_},
                                          quantiser,
                                          dh::ToSpan(ft),
                                          cuts_.cut_ptrs_.ConstDeviceSpan(),
                                          cuts_.cut_values_.ConstDeviceSpan(),
                                          cuts_.min_vals_.ConstDeviceSpan(),
                                          false};
  auto split = evaluator.EvaluateSingleSplit(input, shared_inputs).split;
  ASSERT_NEAR(split.loss_chg, best_score_, 1e-2);
}
}  // namespace tree
}  // namespace xgboost
