#include "hip/hip_runtime.h"

/*!
 * Copyright 2017 XGBoost contributors
 */
#include <xgboost/c_api.h>
#include <xgboost/predictor.h>
#include "gtest/gtest.h"
#include "../helpers.h"

namespace xgboost {
namespace predictor {
TEST(gpu_predictor, Test) {
  std::unique_ptr<Predictor> gpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("gpu_predictor"));
  std::unique_ptr<Predictor> cpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("cpu_predictor"));

  std::vector<std::unique_ptr<RegTree>> trees;
  trees.push_back(std::unique_ptr<RegTree>());
  trees.back()->InitModel();
  (*trees.back())[0].set_leaf(1.5f);
  gbm::GBTreeModel model(0.5);
  model.CommitModel(std::move(trees), 0);
  model.param.num_output_group = 1;

  int n_row = 5;
  int n_col = 5;

  auto dmat = CreateDMatrix(n_row, n_col, 0);

  // Test predict batch
  std::vector<float> gpu_out_predictions;
  std::vector<float> cpu_out_predictions;
  gpu_predictor->PredictBatch(dmat.get(), &gpu_out_predictions, model, 0);
  cpu_predictor->PredictBatch(dmat.get(), &cpu_out_predictions, model, 0);
  float abs_tolerance = 0.001;
  for (int i = 0; i < gpu_out_predictions.size(); i++) {
    ASSERT_LT(std::abs(gpu_out_predictions[i] - cpu_out_predictions[i]),
              abs_tolerance);
  }

  // Test predict instance
  auto batch = dmat->RowIterator()->Value();
  for (int i = 0; i < batch.size; i++) {
    std::vector<float> gpu_instance_out_predictions;
    std::vector<float> cpu_instance_out_predictions;
    cpu_predictor->PredictInstance(batch[i], &cpu_instance_out_predictions,
                                   model);
    gpu_predictor->PredictInstance(batch[i], &gpu_instance_out_predictions,
                                   model);
    ASSERT_EQ(gpu_instance_out_predictions[0], cpu_instance_out_predictions[0]);
  }

  // Test predict leaf
  std::vector<float> gpu_leaf_out_predictions;
  std::vector<float> cpu_leaf_out_predictions;
  cpu_predictor->PredictLeaf(dmat.get(), &cpu_leaf_out_predictions, model);
  gpu_predictor->PredictLeaf(dmat.get(), &gpu_leaf_out_predictions, model);
  for (int i = 0; i < gpu_leaf_out_predictions.size(); i++) {
    ASSERT_EQ(gpu_leaf_out_predictions[i], cpu_leaf_out_predictions[i]);
  }

  // Test predict contribution
  std::vector<float> gpu_out_contribution;
  std::vector<float> cpu_out_contribution;
  cpu_predictor->PredictContribution(dmat.get(), &cpu_out_contribution, model);
  gpu_predictor->PredictContribution(dmat.get(), &gpu_out_contribution, model);
  for (int i = 0; i < gpu_out_contribution.size(); i++) {
    ASSERT_EQ(gpu_out_contribution[i], cpu_out_contribution[i]);
  }
}
}  // namespace predictor
}  // namespace xgboost
