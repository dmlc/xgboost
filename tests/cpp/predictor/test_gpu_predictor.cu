
/*!
 * Copyright 2017 XGBoost contributors
 */
#include <dmlc/logging.h>
#include <dmlc/filesystem.h>
#include <xgboost/c_api.h>
#include <xgboost/predictor.h>
#include <string>
#include "gtest/gtest.h"
#include "../helpers.h"

namespace {

inline void CheckCAPICall(int ret) {
  ASSERT_EQ(ret, 0) << XGBGetLastError();
}

}  // namespace anonymous

extern const std::map<std::string, std::string>&
QueryBoosterConfigurationArguments(BoosterHandle handle);

namespace xgboost {
namespace predictor {

TEST(gpu_predictor, Test) {
  std::unique_ptr<Predictor> gpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("gpu_predictor"));
  std::unique_ptr<Predictor> cpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("cpu_predictor"));

  gpu_predictor->Init({}, {});
  cpu_predictor->Init({}, {});

  std::vector<std::unique_ptr<RegTree>> trees;
  trees.push_back(std::unique_ptr<RegTree>(new RegTree()));
  trees.back()->InitModel();
  (*trees.back())[0].SetLeaf(1.5f);
  (*trees.back()).Stat(0).sum_hess = 1.0f;
  gbm::GBTreeModel model(0.5);
  model.CommitModel(std::move(trees), 0);
  model.param.num_output_group = 1;

  int n_row = 5;
  int n_col = 5;

  auto dmat = CreateDMatrix(n_row, n_col, 0);

  // Test predict batch
  HostDeviceVector<float> gpu_out_predictions;
  HostDeviceVector<float> cpu_out_predictions;
  gpu_predictor->PredictBatch((*dmat).get(), &gpu_out_predictions, model, 0);
  cpu_predictor->PredictBatch((*dmat).get(), &cpu_out_predictions, model, 0);
  std::vector<float>& gpu_out_predictions_h = gpu_out_predictions.HostVector();
  std::vector<float>& cpu_out_predictions_h = cpu_out_predictions.HostVector();
  float abs_tolerance = 0.001;
  for (int i = 0; i < gpu_out_predictions.Size(); i++) {
    ASSERT_NEAR(gpu_out_predictions_h[i], cpu_out_predictions_h[i], abs_tolerance);
  }
  // Test predict instance
  const auto &batch = *(*dmat)->GetRowBatches().begin();
  for (int i = 0; i < batch.Size(); i++) {
    std::vector<float> gpu_instance_out_predictions;
    std::vector<float> cpu_instance_out_predictions;
    cpu_predictor->PredictInstance(batch[i], &cpu_instance_out_predictions,
                                   model);
    gpu_predictor->PredictInstance(batch[i], &gpu_instance_out_predictions,
                                   model);
    ASSERT_EQ(gpu_instance_out_predictions[0], cpu_instance_out_predictions[0]);
  }

  // Test predict leaf
  std::vector<float> gpu_leaf_out_predictions;
  std::vector<float> cpu_leaf_out_predictions;
  cpu_predictor->PredictLeaf((*dmat).get(), &cpu_leaf_out_predictions, model);
  gpu_predictor->PredictLeaf((*dmat).get(), &gpu_leaf_out_predictions, model);
  for (int i = 0; i < gpu_leaf_out_predictions.size(); i++) {
    ASSERT_EQ(gpu_leaf_out_predictions[i], cpu_leaf_out_predictions[i]);
  }

  // Test predict contribution
  std::vector<float> gpu_out_contribution;
  std::vector<float> cpu_out_contribution;
  cpu_predictor->PredictContribution((*dmat).get(), &cpu_out_contribution, model);
  gpu_predictor->PredictContribution((*dmat).get(), &gpu_out_contribution, model);
  for (int i = 0; i < gpu_out_contribution.size(); i++) {
    ASSERT_EQ(gpu_out_contribution[i], cpu_out_contribution[i]);
  }

  delete dmat;
}

// Test whether pickling preserves predictor parameters
TEST(gpu_predictor, MGPU_PicklingTest) {
  int ngpu;
  dh::safe_cuda(hipGetDeviceCount(&ngpu));

  dmlc::TemporaryDirectory tempdir;
  const std::string tmp_file = tempdir.path + "/simple.libsvm";
  CreateBigTestData(tmp_file, 600);

  DMatrixHandle dmat[1];
  BoosterHandle bst, bst2;
  std::vector<bst_float> label;
  for (int i = 0; i < 200; ++i) {
    label.push_back((i % 2 ? 1 : 0));
  }

  // Load data matrix
  CheckCAPICall(XGDMatrixCreateFromFile(tmp_file.c_str(), 0, &dmat[0]));
  CheckCAPICall(XGDMatrixSetFloatInfo(dmat[0], "label", label.data(), 200));
  // Create booster
  CheckCAPICall(XGBoosterCreate(dmat, 1, &bst));
  // Set parameters
  CheckCAPICall(XGBoosterSetParam(bst, "seed", "0"));
  CheckCAPICall(XGBoosterSetParam(bst, "base_score", "0.5"));
  CheckCAPICall(XGBoosterSetParam(bst, "booster", "gbtree"));
  CheckCAPICall(XGBoosterSetParam(bst, "learning_rate", "0.01"));
  CheckCAPICall(XGBoosterSetParam(bst, "max_depth", "8"));
  CheckCAPICall(XGBoosterSetParam(bst, "objective", "binary:logistic"));
  CheckCAPICall(XGBoosterSetParam(bst, "seed", "123"));
  CheckCAPICall(XGBoosterSetParam(bst, "tree_method", "gpu_hist"));
  CheckCAPICall(XGBoosterSetParam(bst, "n_gpus", std::to_string(ngpu).c_str()));
  CheckCAPICall(XGBoosterSetParam(bst, "predictor", "gpu_predictor"));

  // Run boosting iterations
  for (int i = 0; i < 10; ++i) {
    CheckCAPICall(XGBoosterUpdateOneIter(bst, i, dmat[0]));
  }

  // Delete matrix
  CheckCAPICall(XGDMatrixFree(dmat[0]));

  // Pickle
  const char* dptr;
  bst_ulong len;
  std::string buf;
  CheckCAPICall(XGBoosterGetModelRaw(bst, &len, &dptr));
  buf = std::string(dptr, len);
  CheckCAPICall(XGBoosterFree(bst));

  // Unpickle
  CheckCAPICall(XGBoosterCreate(nullptr, 0, &bst2));
  CheckCAPICall(XGBoosterLoadModelFromBuffer(bst2, buf.c_str(), len));

  {  // Query predictor
    const auto& kwargs = QueryBoosterConfigurationArguments(bst2);
    ASSERT_EQ(kwargs.at("predictor"), "gpu_predictor");
    ASSERT_EQ(kwargs.at("n_gpus"), std::to_string(ngpu).c_str());
  }

  {  // Change n_gpus and query again
    CheckCAPICall(XGBoosterSetParam(bst2, "n_gpus", "1"));
    const auto& kwargs = QueryBoosterConfigurationArguments(bst2);
    ASSERT_EQ(kwargs.at("n_gpus"), "1");
  }

  {  // Change predictor and query again
    CheckCAPICall(XGBoosterSetParam(bst2, "predictor", "cpu_predictor"));
    const auto& kwargs = QueryBoosterConfigurationArguments(bst2);
    ASSERT_EQ(kwargs.at("predictor"), "cpu_predictor");
  }

  CheckCAPICall(XGBoosterFree(bst2));
}

// multi-GPU predictor test
TEST(gpu_predictor, MGPU_Test) {
  std::unique_ptr<Predictor> gpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("gpu_predictor"));
  std::unique_ptr<Predictor> cpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("cpu_predictor"));

  gpu_predictor->Init({std::pair<std::string, std::string>("n_gpus", "-1")}, {});
  cpu_predictor->Init({}, {});

  for (size_t i = 1; i < 33; i *= 2) {
    int n_row = i, n_col = i;
    auto dmat = CreateDMatrix(n_row, n_col, 0);

    std::vector<std::unique_ptr<RegTree>> trees;
    trees.push_back(std::unique_ptr<RegTree>(new RegTree()));
    trees.back()->InitModel();
    (*trees.back())[0].SetLeaf(1.5f);
    (*trees.back()).Stat(0).sum_hess = 1.0f;
    gbm::GBTreeModel model(0.5);
    model.CommitModel(std::move(trees), 0);
    model.param.num_output_group = 1;

    // Test predict batch
    HostDeviceVector<float> gpu_out_predictions;
    HostDeviceVector<float> cpu_out_predictions;

    gpu_predictor->PredictBatch((*dmat).get(), &gpu_out_predictions, model, 0);
    cpu_predictor->PredictBatch((*dmat).get(), &cpu_out_predictions, model, 0);

    std::vector<float>& gpu_out_predictions_h = gpu_out_predictions.HostVector();
    std::vector<float>& cpu_out_predictions_h = cpu_out_predictions.HostVector();
    float abs_tolerance = 0.001;
    for (int i = 0; i < gpu_out_predictions.Size(); i++) {
      ASSERT_NEAR(gpu_out_predictions_h[i], cpu_out_predictions_h[i], abs_tolerance);
    }
    delete dmat;
  }
}

}  // namespace predictor
}  // namespace xgboost
