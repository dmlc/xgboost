/*!
 * Copyright 2017-2020 XGBoost contributors
 */
#include <gtest/gtest.h>
#include <dmlc/filesystem.h>
#include <xgboost/c_api.h>
#include <xgboost/predictor.h>
#include <xgboost/logging.h>
#include <xgboost/learner.h>
#include <string>

#include "../helpers.h"
#include "../../../src/gbm/gbtree_model.h"
#include "../../../src/data/device_adapter.cuh"
#include "test_predictor.h"

namespace xgboost {
namespace predictor {

TEST(GPUPredictor, Basic) {
  auto cpu_lparam = CreateEmptyGenericParam(-1);
  auto gpu_lparam = CreateEmptyGenericParam(0);

  std::unique_ptr<Predictor> gpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("gpu_predictor", &gpu_lparam));
  std::unique_ptr<Predictor> cpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("cpu_predictor", &cpu_lparam));

  gpu_predictor->Configure({});
  cpu_predictor->Configure({});

  for (size_t i = 1; i < 33; i *= 2) {
    int n_row = i, n_col = i;
    auto dmat = RandomDataGenerator(n_row, n_col, 0).GenerateDMatrix();

    LearnerModelParam param;
    param.num_feature = n_col;
    param.num_output_group = 1;
    param.base_score = 0.5;

    gbm::GBTreeModel model = CreateTestModel(&param);

    // Test predict batch
    PredictionCacheEntry gpu_out_predictions;
    PredictionCacheEntry cpu_out_predictions;

    gpu_predictor->PredictBatch(dmat.get(), &gpu_out_predictions, model, 0);
    ASSERT_EQ(model.trees.size(), gpu_out_predictions.version);
    cpu_predictor->PredictBatch(dmat.get(), &cpu_out_predictions, model, 0);

    std::vector<float>& gpu_out_predictions_h = gpu_out_predictions.predictions.HostVector();
    std::vector<float>& cpu_out_predictions_h = cpu_out_predictions.predictions.HostVector();
    float abs_tolerance = 0.001;
    for (int j = 0; j < gpu_out_predictions.predictions.Size(); j++) {
      ASSERT_NEAR(gpu_out_predictions_h[j], cpu_out_predictions_h[j], abs_tolerance);
    }
  }
}

TEST(GPUPredictor, EllpackBasic) {
  size_t constexpr kCols {8};
  for (size_t bins = 2; bins < 258; bins += 16) {
    size_t rows = bins * 16;
    auto p_m = RandomDataGenerator{rows, kCols, 0.0}
         .Bins(bins)
         .Device(0)
         .GenerateDeviceDMatrix(true);
    TestPredictionFromGradientIndex<EllpackPage>("gpu_predictor", rows, kCols, p_m);
    TestPredictionFromGradientIndex<EllpackPage>("gpu_predictor", bins, kCols, p_m);
  }
}

TEST(GPUPredictor, EllpackTraining) {
  size_t constexpr kRows { 128 }, kCols { 16 }, kBins { 64 };
  auto p_ellpack = RandomDataGenerator{kRows, kCols, 0.0}
       .Bins(kBins)
       .Device(0)
       .GenerateDeviceDMatrix(true);
  HostDeviceVector<float> storage(kRows * kCols);
  auto columnar = RandomDataGenerator{kRows, kCols, 0.0}
       .Device(0)
       .GenerateArrayInterface(&storage);
  auto adapter = data::CupyAdapter(columnar);
  std::shared_ptr<DMatrix> p_full {
    DMatrix::Create(&adapter, std::numeric_limits<float>::quiet_NaN(), 1)
  };
  TestTrainingPrediction(kRows, kBins, "gpu_hist", p_full, p_ellpack);
}

TEST(GPUPredictor, ExternalMemoryTest) {
  auto lparam = CreateEmptyGenericParam(0);
  std::unique_ptr<Predictor> gpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("gpu_predictor", &lparam));
  gpu_predictor->Configure({});

  LearnerModelParam param;
  param.num_feature = 5;
  const int n_classes = 3;
  param.num_output_group = n_classes;
  param.base_score = 0.5;

  gbm::GBTreeModel model = CreateTestModel(&param, n_classes);
  std::vector<std::unique_ptr<DMatrix>> dmats;
  dmlc::TemporaryDirectory tmpdir;
  std::string file0 = tmpdir.path + "/big_0.libsvm";
  std::string file1 = tmpdir.path + "/big_1.libsvm";
  std::string file2 = tmpdir.path + "/big_2.libsvm";
  dmats.push_back(CreateSparsePageDMatrix(400, 64UL, file0));
  dmats.push_back(CreateSparsePageDMatrix(800, 128UL, file1));
  dmats.push_back(CreateSparsePageDMatrix(8000, 1024UL, file2));

  for (const auto& dmat: dmats) {
    dmat->Info().base_margin_.Resize(dmat->Info().num_row_ * n_classes, 0.5);
    PredictionCacheEntry out_predictions;
    gpu_predictor->PredictBatch(dmat.get(), &out_predictions, model, 0);
    EXPECT_EQ(out_predictions.predictions.Size(), dmat->Info().num_row_ * n_classes);
    const std::vector<float> &host_vector = out_predictions.predictions.ConstHostVector();
    for (int i = 0; i < host_vector.size() / n_classes; i++) {
      ASSERT_EQ(host_vector[i * n_classes], 2.0);
      ASSERT_EQ(host_vector[i * n_classes + 1], 0.5);
      ASSERT_EQ(host_vector[i * n_classes + 2], 0.5);
    }
  }
}

TEST(GPUPredictor, InplacePredictCupy) {
  size_t constexpr kRows{128}, kCols{64};
  RandomDataGenerator gen(kRows, kCols, 0.5);
  gen.Device(0);
  HostDeviceVector<float> data;
  std::string interface_str = gen.GenerateArrayInterface(&data);
  auto x = std::make_shared<data::CupyAdapter>(interface_str);
  TestInplacePrediction(x, "gpu_predictor", kRows, kCols, 0);
}

TEST(GPUPredictor, InplacePredictCuDF) {
  size_t constexpr kRows{128}, kCols{64};
  RandomDataGenerator gen(kRows, kCols, 0.5);
  gen.Device(0);
  std::vector<HostDeviceVector<float>> storage(kCols);
  auto interface_str = gen.GenerateColumnarArrayInterface(&storage);
  auto x = std::make_shared<data::CudfAdapter>(interface_str);
  TestInplacePrediction(x, "gpu_predictor", kRows, kCols, 0);
}

TEST(GPUPredictor, MGPU_InplacePredict) {  // NOLINT
  int32_t n_gpus = xgboost::common::AllVisibleGPUs();
  if (n_gpus <= 1) {
    LOG(WARNING) << "GPUPredictor.MGPU_InplacePredict is skipped.";
    return;
  }
  size_t constexpr kRows{128}, kCols{64};
  RandomDataGenerator gen(kRows, kCols, 0.5);
  gen.Device(1);
  HostDeviceVector<float> data;
  std::string interface_str = gen.GenerateArrayInterface(&data);
  auto x = std::make_shared<data::CupyAdapter>(interface_str);
  TestInplacePrediction(x, "gpu_predictor", kRows, kCols, 1);
  EXPECT_THROW(TestInplacePrediction(x, "gpu_predictor", kRows, kCols, 0),
               dmlc::Error);
}

TEST(GpuPredictor, LesserFeatures) {
  TestPredictionWithLesserFeatures("gpu_predictor");
}
// Very basic test of empty model
TEST(GPUPredictor, ShapStump) {
  hipSetDevice(0);
  LearnerModelParam param;
  param.num_feature = 1;
  param.num_output_group = 1;
  param.base_score = 0.5;
  gbm::GBTreeModel model(&param);
  std::vector<std::unique_ptr<RegTree>> trees;
  trees.push_back(std::unique_ptr<RegTree>(new RegTree));
  model.CommitModel(std::move(trees), 0);

  auto gpu_lparam = CreateEmptyGenericParam(0);
  std::unique_ptr<Predictor> gpu_predictor = std::unique_ptr<Predictor>(
      Predictor::Create("gpu_predictor", &gpu_lparam));
  gpu_predictor->Configure({});
  HostDeviceVector<float> predictions;
  auto dmat = RandomDataGenerator(3, 1, 0).GenerateDMatrix();
  gpu_predictor->PredictContribution(dmat.get(), &predictions, model);
  auto& phis = predictions.HostVector();
  EXPECT_EQ(phis[0], 0.0);
  EXPECT_EQ(phis[1], param.base_score);
  EXPECT_EQ(phis[2], 0.0);
  EXPECT_EQ(phis[3], param.base_score);
  EXPECT_EQ(phis[4], 0.0);
  EXPECT_EQ(phis[5], param.base_score);
}

TEST(GPUPredictor, Shap) {
  LearnerModelParam param;
  param.num_feature = 1;
  param.num_output_group = 1;
  param.base_score = 0.5;
  gbm::GBTreeModel model(&param);
  std::vector<std::unique_ptr<RegTree>> trees;
  trees.push_back(std::unique_ptr<RegTree>(new RegTree));
  trees[0]->ExpandNode(0, 0, 0.5, true, 1.0, -1.0, 1.0, 0.0, 5.0, 2.0, 3.0);
  model.CommitModel(std::move(trees), 0);

  auto gpu_lparam = CreateEmptyGenericParam(0);
  auto cpu_lparam = CreateEmptyGenericParam(-1);
  std::unique_ptr<Predictor> gpu_predictor = std::unique_ptr<Predictor>(
      Predictor::Create("gpu_predictor", &gpu_lparam));
  std::unique_ptr<Predictor> cpu_predictor = std::unique_ptr<Predictor>(
      Predictor::Create("cpu_predictor", &cpu_lparam));
  gpu_predictor->Configure({});
  cpu_predictor->Configure({});
  HostDeviceVector<float> predictions;
  HostDeviceVector<float> cpu_predictions;
  auto dmat = RandomDataGenerator(3, 1, 0).GenerateDMatrix();
  gpu_predictor->PredictContribution(dmat.get(), &predictions, model);
  cpu_predictor->PredictContribution(dmat.get(), &cpu_predictions, model);
  auto& phis = predictions.HostVector();
  auto& cpu_phis = cpu_predictions.HostVector();
  for (auto i = 0ull; i < phis.size(); i++) {
    EXPECT_NEAR(cpu_phis[i], phis[i], 1e-3);
  }
}

TEST(GPUPredictor, CategoricalPrediction) {
  TestCategoricalPrediction("gpu_predictor");
}

TEST(GPUPredictor, PredictLeafBasic) {
  size_t constexpr kRows = 5, kCols = 5;
  auto dmat = RandomDataGenerator(kRows, kCols, 0).Device(0).GenerateDMatrix();
  auto lparam = CreateEmptyGenericParam(GPUIDX);
  std::unique_ptr<Predictor> gpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("gpu_predictor", &lparam));
  gpu_predictor->Configure({});

  LearnerModelParam param;
  param.num_feature = kCols;
  param.base_score = 0.0;
  param.num_output_group = 1;

  gbm::GBTreeModel model = CreateTestModel(&param);

  HostDeviceVector<float> leaf_out_predictions;
  gpu_predictor->PredictLeaf(dmat.get(), &leaf_out_predictions, model);
  auto const& h_leaf_out_predictions = leaf_out_predictions.ConstHostVector();
  for (auto v : h_leaf_out_predictions) {
    ASSERT_EQ(v, 0);
  }
}
}  // namespace predictor
}  // namespace xgboost
