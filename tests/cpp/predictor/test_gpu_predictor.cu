
/*!
 * Copyright 2017-2019 XGBoost contributors
 */
#include <dmlc/logging.h>
#include <dmlc/filesystem.h>
#include <xgboost/c_api.h>
#include <xgboost/predictor.h>
#include <string>
#include "gtest/gtest.h"
#include "../helpers.h"

namespace {

inline void CheckCAPICall(int ret) {
  ASSERT_EQ(ret, 0) << XGBGetLastError();
}

}  // namespace anonymous

extern const std::map<std::string, std::string>&
QueryBoosterConfigurationArguments(BoosterHandle handle);

namespace xgboost {
namespace predictor {

TEST(gpu_predictor, Test) {
  auto cpu_lparam = CreateEmptyGenericParam(0, 0);
  auto gpu_lparam = CreateEmptyGenericParam(0, 1);

  std::unique_ptr<Predictor> gpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("gpu_predictor", &gpu_lparam));
  std::unique_ptr<Predictor> cpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("cpu_predictor", &cpu_lparam));

  gpu_predictor->Init({}, {});
  cpu_predictor->Init({}, {});

  gbm::GBTreeModel model = CreateTestModel();

  int n_row = 5;
  int n_col = 5;

  auto dmat = CreateDMatrix(n_row, n_col, 0);

  // Test predict batch
  HostDeviceVector<float> gpu_out_predictions;
  HostDeviceVector<float> cpu_out_predictions;
  gpu_predictor->PredictBatch((*dmat).get(), &gpu_out_predictions, model, 0);
  cpu_predictor->PredictBatch((*dmat).get(), &cpu_out_predictions, model, 0);
  std::vector<float>& gpu_out_predictions_h = gpu_out_predictions.HostVector();
  std::vector<float>& cpu_out_predictions_h = cpu_out_predictions.HostVector();
  float abs_tolerance = 0.001;
  for (int i = 0; i < gpu_out_predictions.Size(); i++) {
    ASSERT_NEAR(gpu_out_predictions_h[i], cpu_out_predictions_h[i], abs_tolerance);
  }
  // Test predict instance
  const auto &batch = *(*dmat)->GetRowBatches().begin();
  for (int i = 0; i < batch.Size(); i++) {
    std::vector<float> gpu_instance_out_predictions;
    std::vector<float> cpu_instance_out_predictions;
    cpu_predictor->PredictInstance(batch[i], &cpu_instance_out_predictions,
                                   model);
    gpu_predictor->PredictInstance(batch[i], &gpu_instance_out_predictions,
                                   model);
    ASSERT_EQ(gpu_instance_out_predictions[0], cpu_instance_out_predictions[0]);
  }

  // Test predict leaf
  std::vector<float> gpu_leaf_out_predictions;
  std::vector<float> cpu_leaf_out_predictions;
  cpu_predictor->PredictLeaf((*dmat).get(), &cpu_leaf_out_predictions, model);
  gpu_predictor->PredictLeaf((*dmat).get(), &gpu_leaf_out_predictions, model);
  for (int i = 0; i < gpu_leaf_out_predictions.size(); i++) {
    ASSERT_EQ(gpu_leaf_out_predictions[i], cpu_leaf_out_predictions[i]);
  }

  // Test predict contribution
  std::vector<float> gpu_out_contribution;
  std::vector<float> cpu_out_contribution;
  cpu_predictor->PredictContribution((*dmat).get(), &cpu_out_contribution, model);
  gpu_predictor->PredictContribution((*dmat).get(), &gpu_out_contribution, model);
  for (int i = 0; i < gpu_out_contribution.size(); i++) {
    ASSERT_EQ(gpu_out_contribution[i], cpu_out_contribution[i]);
  }

  delete dmat;
}

TEST(gpu_predictor, ExternalMemoryTest) {
  auto lparam = CreateEmptyGenericParam(0, 1);
  std::unique_ptr<Predictor> gpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("gpu_predictor", &lparam));
  gpu_predictor->Init({}, {});
  gbm::GBTreeModel model = CreateTestModel();
  std::unique_ptr<DMatrix> dmat = CreateSparsePageDMatrix(32, 64);

  // Test predict batch
  HostDeviceVector<float> out_predictions;
  gpu_predictor->PredictBatch(dmat.get(), &out_predictions, model, 0);
  EXPECT_EQ(out_predictions.Size(), dmat->Info().num_row_);
  for (const auto& v : out_predictions.HostVector()) {
    ASSERT_EQ(v, 1.5);
  }

  // Test predict leaf
  std::vector<float> leaf_out_predictions;
  gpu_predictor->PredictLeaf(dmat.get(), &leaf_out_predictions, model);
  EXPECT_EQ(leaf_out_predictions.size(), dmat->Info().num_row_);
  for (const auto& v : leaf_out_predictions) {
    ASSERT_EQ(v, 0);
  }

  // Test predict contribution
  std::vector<float> out_contribution;
  gpu_predictor->PredictContribution(dmat.get(), &out_contribution, model);
  EXPECT_EQ(out_contribution.size(), dmat->Info().num_row_);
  for (const auto& v : out_contribution) {
    ASSERT_EQ(v, 1.5);
  }

  // Test predict contribution (approximate method)
  std::vector<float> out_contribution_approximate;
  gpu_predictor->PredictContribution(dmat.get(), &out_contribution_approximate, model, true);
  EXPECT_EQ(out_contribution_approximate.size(), dmat->Info().num_row_);
  for (const auto& v : out_contribution_approximate) {
    ASSERT_EQ(v, 1.5);
  }
}

#if defined(XGBOOST_USE_NCCL)
// Test whether pickling preserves predictor parameters
TEST(gpu_predictor, MGPU_PicklingTest) {
  int const ngpu = GPUSet::AllVisible().Size();

  dmlc::TemporaryDirectory tempdir;
  const std::string tmp_file = tempdir.path + "/simple.libsvm";
  CreateBigTestData(tmp_file, 600);

  DMatrixHandle dmat[1];
  BoosterHandle bst, bst2;
  std::vector<bst_float> label;
  for (int i = 0; i < 200; ++i) {
    label.push_back((i % 2 ? 1 : 0));
  }

  // Load data matrix
  CheckCAPICall(XGDMatrixCreateFromFile(tmp_file.c_str(), 0, &dmat[0]));
  CheckCAPICall(XGDMatrixSetFloatInfo(dmat[0], "label", label.data(), 200));
  // Create booster
  CheckCAPICall(XGBoosterCreate(dmat, 1, &bst));
  // Set parameters
  CheckCAPICall(XGBoosterSetParam(bst, "seed", "0"));
  CheckCAPICall(XGBoosterSetParam(bst, "base_score", "0.5"));
  CheckCAPICall(XGBoosterSetParam(bst, "booster", "gbtree"));
  CheckCAPICall(XGBoosterSetParam(bst, "learning_rate", "0.01"));
  CheckCAPICall(XGBoosterSetParam(bst, "max_depth", "8"));
  CheckCAPICall(XGBoosterSetParam(bst, "objective", "binary:logistic"));
  CheckCAPICall(XGBoosterSetParam(bst, "seed", "123"));
  CheckCAPICall(XGBoosterSetParam(bst, "tree_method", "gpu_hist"));
  CheckCAPICall(XGBoosterSetParam(bst, "n_gpus", std::to_string(ngpu).c_str()));
  CheckCAPICall(XGBoosterSetParam(bst, "predictor", "gpu_predictor"));

  // Run boosting iterations
  for (int i = 0; i < 10; ++i) {
    CheckCAPICall(XGBoosterUpdateOneIter(bst, i, dmat[0]));
  }

  // Delete matrix
  CheckCAPICall(XGDMatrixFree(dmat[0]));

  // Pickle
  const char* dptr;
  bst_ulong len;
  std::string buf;
  CheckCAPICall(XGBoosterGetModelRaw(bst, &len, &dptr));
  buf = std::string(dptr, len);
  CheckCAPICall(XGBoosterFree(bst));

  // Unpickle
  CheckCAPICall(XGBoosterCreate(nullptr, 0, &bst2));
  CheckCAPICall(XGBoosterLoadModelFromBuffer(bst2, buf.c_str(), len));

  {  // Query predictor
    const auto& kwargs = QueryBoosterConfigurationArguments(bst2);
    ASSERT_EQ(kwargs.at("predictor"), "gpu_predictor");
    ASSERT_EQ(kwargs.at("n_gpus"), std::to_string(ngpu).c_str());
  }

  {  // Change n_gpus and query again
    CheckCAPICall(XGBoosterSetParam(bst2, "n_gpus", "1"));
    const auto& kwargs = QueryBoosterConfigurationArguments(bst2);
    ASSERT_EQ(kwargs.at("n_gpus"), "1");
  }

  {  // Change predictor and query again
    CheckCAPICall(XGBoosterSetParam(bst2, "predictor", "cpu_predictor"));
    const auto& kwargs = QueryBoosterConfigurationArguments(bst2);
    ASSERT_EQ(kwargs.at("predictor"), "cpu_predictor");
  }

  CheckCAPICall(XGBoosterFree(bst2));
}

// multi-GPU predictor test
TEST(gpu_predictor, MGPU_Test) {
  auto cpu_lparam = CreateEmptyGenericParam(0, 0);
  auto gpu_lparam = CreateEmptyGenericParam(0, -1);

  std::unique_ptr<Predictor> gpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("gpu_predictor", &gpu_lparam));
  std::unique_ptr<Predictor> cpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("cpu_predictor", &cpu_lparam));

  cpu_predictor->Init({}, {});

  for (size_t i = 1; i < 33; i *= 2) {
    int n_row = i, n_col = i;
    auto dmat = CreateDMatrix(n_row, n_col, 0);

    gbm::GBTreeModel model = CreateTestModel();

    // Test predict batch
    HostDeviceVector<float> gpu_out_predictions;
    HostDeviceVector<float> cpu_out_predictions;

    gpu_predictor->PredictBatch((*dmat).get(), &gpu_out_predictions, model, 0);
    cpu_predictor->PredictBatch((*dmat).get(), &cpu_out_predictions, model, 0);

    std::vector<float>& gpu_out_predictions_h = gpu_out_predictions.HostVector();
    std::vector<float>& cpu_out_predictions_h = cpu_out_predictions.HostVector();
    float abs_tolerance = 0.001;
    for (int j = 0; j < gpu_out_predictions.Size(); j++) {
      ASSERT_NEAR(gpu_out_predictions_h[j], cpu_out_predictions_h[j], abs_tolerance);
    }
    delete dmat;
  }
}

// multi-GPU predictor external memory test
TEST(gpu_predictor, MGPU_ExternalMemoryTest) {
  auto gpu_lparam = CreateEmptyGenericParam(0, -1);

  std::unique_ptr<Predictor> gpu_predictor =
      std::unique_ptr<Predictor>(Predictor::Create("gpu_predictor", &gpu_lparam));
  gpu_predictor->Init({}, {});

  gbm::GBTreeModel model = CreateTestModel();
  const int n_classes = 3;
  model.param.num_output_group = n_classes;
  std::vector<std::unique_ptr<DMatrix>> dmats;
  dmats.push_back(CreateSparsePageDMatrix(9, 64UL));
  dmats.push_back(CreateSparsePageDMatrix(128, 128UL));
  dmats.push_back(CreateSparsePageDMatrix(1024, 1024UL));

  for (const auto& dmat: dmats) {
    // Test predict batch
    HostDeviceVector<float> out_predictions;
    gpu_predictor->PredictBatch(dmat.get(), &out_predictions, model, 0);
    EXPECT_EQ(out_predictions.Size(), dmat->Info().num_row_ * n_classes);
    const std::vector<float> &host_vector = out_predictions.ConstHostVector();
    for (int i = 0; i < host_vector.size() / n_classes; i++) {
      ASSERT_EQ(host_vector[i * n_classes], 1.5);
      ASSERT_EQ(host_vector[i * n_classes + 1], 0.);
      ASSERT_EQ(host_vector[i * n_classes + 2], 0.);
    }
  }
}
#endif  // defined(XGBOOST_USE_NCCL)
}  // namespace predictor
}  // namespace xgboost
