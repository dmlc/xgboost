#include "hip/hip_runtime.h"
/*!
 * Copyright (c) by Contributors 2020
 */
#include <gtest/gtest.h>
#include "test_survival_metric.h"
#include "xgboost/metric.h"

/** Tests for Survival metrics that should run both on CPU and GPU **/

namespace xgboost {
namespace common {
TEST(Metric, DeclareUnifiedTest(AFTNegLogLik)) { VerifyAFTNegLogLik(); }

TEST_F(DeclareUnifiedDistributedTest(MetricTest), AFTNegLogLikRowSplit) {
  RunWithInMemoryCommunicator(world_size_, &VerifyAFTNegLogLik, DataSplitMode::kRow);
}

TEST_F(DeclareUnifiedDistributedTest(MetricTest), AFTNegLogLikColumnSplit) {
  RunWithInMemoryCommunicator(world_size_, &VerifyAFTNegLogLik, DataSplitMode::kCol);
}

TEST(Metric, DeclareUnifiedTest(IntervalRegressionAccuracy)) { VerifyIntervalRegressionAccuracy(); }

TEST_F(DeclareUnifiedDistributedTest(MetricTest), IntervalRegressionAccuracyRowSplit) {
  RunWithInMemoryCommunicator(world_size_, &VerifyIntervalRegressionAccuracy, DataSplitMode::kRow);
}

TEST_F(DeclareUnifiedDistributedTest(MetricTest), IntervalRegressionAccuracyColumnSplit) {
  RunWithInMemoryCommunicator(world_size_, &VerifyIntervalRegressionAccuracy, DataSplitMode::kCol);
}

// Test configuration of AFT metric
TEST(AFTNegLogLikMetric, DeclareUnifiedTest(Configuration)) {
  auto ctx = xgboost::CreateEmptyGenericParam(GPUIDX);
  std::unique_ptr<Metric> metric(Metric::Create("aft-nloglik", &ctx));
  metric->Configure({{"aft_loss_distribution", "normal"}, {"aft_loss_distribution_scale", "10"}});

  // Configuration round-trip test
  Json j_obj{ Object() };
  metric->SaveConfig(&j_obj);
  auto aft_param_json = j_obj["aft_loss_param"];
  EXPECT_EQ(get<String>(aft_param_json["aft_loss_distribution"]), "normal");
  EXPECT_EQ(get<String>(aft_param_json["aft_loss_distribution_scale"]), "10");

  CheckDeterministicMetricElementWise(StringView{"aft-nloglik"}, GPUIDX);
}
}  // namespace common
}  // namespace xgboost
