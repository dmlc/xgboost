
#include <hip/hip_runtime.h>
/**
 * Copyright 2022-2023, XGBoost contributors
 */
#ifdef XGBOOST_USE_NCCL

#include <gtest/gtest.h>

#include <bitset>
#include <string>  // for string

#include "../../../src/collective/comm.cuh"
#include "../../../src/collective/communicator-inl.cuh"
#include "../../../src/collective/nccl_device_communicator.cuh"
#include "../helpers.h"

namespace xgboost {
namespace collective {

TEST(NcclDeviceCommunicatorSimpleTest, ThrowOnInvalidDeviceOrdinal) {
  auto construct = []() { NcclDeviceCommunicator comm{-1, false, DefaultNcclName()}; };
  EXPECT_THROW(construct(), dmlc::Error);
}

TEST(NcclDeviceCommunicatorSimpleTest, SystemError) {
  auto stub = std::make_shared<NcclStub>(DefaultNcclName());
  auto rc = stub->GetNcclResult(ncclSystemError);
  auto msg = rc.Report();
  ASSERT_TRUE(msg.find("environment variables") != std::string::npos);
}

namespace {
void VerifyAllReduceBitwiseAND() {
  auto const rank = collective::GetRank();
  std::bitset<64> original{};
  original[rank] = true;
  HostDeviceVector<uint64_t> buffer({original.to_ullong()}, DeviceOrd::CUDA(rank));
  collective::AllReduce<collective::Operation::kBitwiseAND>(rank, buffer.DevicePointer(), 1);
  collective::Synchronize(rank);
  EXPECT_EQ(buffer.HostVector()[0], 0ULL);
}
}  // anonymous namespace

TEST(NcclDeviceCommunicator, MGPUAllReduceBitwiseAND) {
  auto const n_gpus = common::AllVisibleGPUs();
  if (n_gpus <= 1) {
    GTEST_SKIP() << "Skipping MGPUAllReduceBitwiseAND test with # GPUs = " << n_gpus;
  }
  auto constexpr kUseNccl = true;
  RunWithInMemoryCommunicator<kUseNccl>(n_gpus, VerifyAllReduceBitwiseAND);
}

namespace {
void VerifyAllReduceBitwiseOR() {
  auto const world_size = collective::GetWorldSize();
  auto const rank = collective::GetRank();
  std::bitset<64> original{};
  original[rank] = true;
  HostDeviceVector<uint64_t> buffer({original.to_ullong()}, DeviceOrd::CUDA(rank));
  collective::AllReduce<collective::Operation::kBitwiseOR>(rank, buffer.DevicePointer(), 1);
  collective::Synchronize(rank);
  EXPECT_EQ(buffer.HostVector()[0], (1ULL << world_size) - 1);
}
}  // anonymous namespace

TEST(NcclDeviceCommunicator, MGPUAllReduceBitwiseOR) {
  auto const n_gpus = common::AllVisibleGPUs();
  if (n_gpus <= 1) {
    GTEST_SKIP() << "Skipping MGPUAllReduceBitwiseOR test with # GPUs = " << n_gpus;
  }
  auto constexpr kUseNccl = true;
  RunWithInMemoryCommunicator<kUseNccl>(n_gpus, VerifyAllReduceBitwiseOR);
}

namespace {
void VerifyAllReduceBitwiseXOR() {
  auto const world_size = collective::GetWorldSize();
  auto const rank = collective::GetRank();
  std::bitset<64> original{~0ULL};
  original[rank] = false;
  HostDeviceVector<uint64_t> buffer({original.to_ullong()}, DeviceOrd::CUDA(rank));
  collective::AllReduce<collective::Operation::kBitwiseXOR>(rank, buffer.DevicePointer(), 1);
  collective::Synchronize(rank);
  EXPECT_EQ(buffer.HostVector()[0], (1ULL << world_size) - 1);
}
}  // anonymous namespace

TEST(NcclDeviceCommunicator, MGPUAllReduceBitwiseXOR) {
  auto const n_gpus = common::AllVisibleGPUs();
  if (n_gpus <= 1) {
    GTEST_SKIP() << "Skipping MGPUAllReduceBitwiseXOR test with # GPUs = " << n_gpus;
  }
  auto constexpr kUseNccl = true;
  RunWithInMemoryCommunicator<kUseNccl>(n_gpus, VerifyAllReduceBitwiseXOR);
}

}  // namespace collective
}  // namespace xgboost

#endif  // XGBOOST_USE_NCCL
