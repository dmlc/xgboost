
#include <hip/hip_runtime.h>
/**
 * Copyright 2023, XGBoost Contributors
 */
#if defined(XGBOOST_USE_NCCL)
#include <gtest/gtest.h>
#include <thrust/host_vector.h>  // for host_vector

#include "../../../src/common/common.h"
#include "../../../src/common/device_helpers.cuh"  // for ToSpan,  device_vector
#include "../../../src/common/type.h"              // for EraseType
#include "test_worker.cuh"                         // for NCCLWorkerForTest
#include "test_worker.h"                           // for WorkerForTest, TestDistributed

namespace xgboost::collective {
namespace {
class MGPUAllreduceTest : public SocketTest {};

class Worker : public NCCLWorkerForTest {
 public:
  using NCCLWorkerForTest::NCCLWorkerForTest;

  void BitOr() {
    dh::device_vector<std::uint32_t> data(comm_.World(), 0);
    data[comm_.Rank()] = ~std::uint32_t{0};
    auto rc = nccl_coll_->Allreduce(*nccl_comm_, common::EraseType(dh::ToSpan(data)),
                                    ArrayInterfaceHandler::kU4, Op::kBitwiseOR);
    ASSERT_TRUE(rc.OK()) << rc.Report();
    thrust::host_vector<std::uint32_t> h_data(data.size());
    thrust::copy(data.cbegin(), data.cend(), h_data.begin());
    for (auto v : h_data) {
      ASSERT_EQ(v, ~std::uint32_t{0});
    }
  }

  void Acc() {
    dh::device_vector<double> data(314, 1.5);
    auto rc = nccl_coll_->Allreduce(*nccl_comm_, common::EraseType(dh::ToSpan(data)),
                                    ArrayInterfaceHandler::kF8, Op::kSum);
    ASSERT_TRUE(rc.OK()) << rc.Report();
    for (std::size_t i = 0; i < data.size(); ++i) {
      auto v = data[i];
      ASSERT_EQ(v, 1.5 * static_cast<double>(comm_.World())) << i;
    }
  }
};
}  // namespace

TEST_F(MGPUAllreduceTest, BitOr) {
  auto n_workers = common::AllVisibleGPUs();
  TestDistributed(n_workers, [=](std::string host, std::int32_t port, std::chrono::seconds timeout,
                                 std::int32_t r) {
    Worker w{host, port, timeout, n_workers, r};
    w.Setup();
    w.BitOr();
  });
}

TEST_F(MGPUAllreduceTest, Sum) {
  auto n_workers = common::AllVisibleGPUs();
  TestDistributed(n_workers, [=](std::string host, std::int32_t port, std::chrono::seconds timeout,
                                 std::int32_t r) {
    Worker w{host, port, timeout, n_workers, r};
    w.Setup();
    w.Acc();
  });
}
}  // namespace xgboost::collective
#endif  // defined(XGBOOST_USE_NCCL)
