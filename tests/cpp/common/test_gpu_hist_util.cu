#include "hip/hip_runtime.h"
#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/hist_util.h"
#include "gtest/gtest.h"
#include "xgboost/c_api.h"
#include <algorithm>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>

namespace xgboost {
namespace common {

TEST(gpu_hist_util, TestDeviceSketch) {
  // create the data
  int nrows = 10001;
  std::vector<float> test_data(nrows);
  auto count_iter = thrust::make_counting_iterator(0);
  // fill in reverse order
  std::copy(count_iter, count_iter + nrows, test_data.rbegin());

  // create the DMatrix
  DMatrixHandle dmat_handle;
  XGDMatrixCreateFromMat(test_data.data(), nrows, 1, -1,
                         &dmat_handle);
  auto dmat = *static_cast<std::shared_ptr<xgboost::DMatrix> *>(dmat_handle);

  // parameters for finding quantiles
  tree::TrainParam p;
  p.max_bin = 20;
  p.gpu_id = 0;
  p.n_gpus = 1;
  // ensure that the exact quantiles are found
  p.gpu_batch_nrows = nrows * 10;

  // find quantiles on the CPU
  HistCutMatrix hmat_cpu;
  hmat_cpu.Init(dmat.get(), p.max_bin);

  // find the cuts on the GPU
  dmlc::DataIter<SparsePage>* iter = dmat->RowIterator();
  iter->BeforeFirst();
  CHECK(iter->Next());
  const SparsePage& batch = iter->Value();
  HistCutMatrix hmat_gpu;
  DeviceSketch(batch, dmat->Info(), p, &hmat_gpu);
  CHECK(!iter->Next());

  // compare the cuts
  double eps = 1e-2;
  ASSERT_EQ(hmat_gpu.min_val.size(), 1);
  ASSERT_EQ(hmat_gpu.row_ptr.size(), 2);
  ASSERT_EQ(hmat_gpu.cut.size(), hmat_cpu.cut.size());
  ASSERT_LT(fabs(hmat_cpu.min_val[0] - hmat_gpu.min_val[0]), eps * nrows);
  for (int i = 0; i < hmat_gpu.cut.size(); ++i) {
    ASSERT_LT(fabs(hmat_cpu.cut[i] - hmat_gpu.cut[i]), eps * nrows);
  }
}

}  // namespace common
}  // namespace xgboost
