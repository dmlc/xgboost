#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>

#include "gtest/gtest.h"

#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>

#include "xgboost/c_api.h"

#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/hist_util.h"
#include "../../../src/tree/updater_gpu_hist.cu"

#include "../helpers.h"

namespace xgboost {
namespace common {

void TestDeviceSketch(const GPUSet& devices, bool use_external_memory = false) {
  // create the data
  int nrows = 10001;
  std::shared_ptr<xgboost::DMatrix> *dmat = nullptr;

  size_t num_cols = 1;
  if (!use_external_memory) {
     std::vector<float> test_data(nrows);
     auto count_iter = thrust::make_counting_iterator(0);
     // fill in reverse order
     std::copy(count_iter, count_iter + nrows, test_data.rbegin());

     // create the DMatrix
     DMatrixHandle dmat_handle;
     XGDMatrixCreateFromMat(test_data.data(), nrows, 1, -1,
                            &dmat_handle);
     dmat = static_cast<std::shared_ptr<xgboost::DMatrix> *>(dmat_handle);
  } else {
     auto sp_dmat = CreateSparsePageDMatrix(nrows * 3, 128UL); // 3 entries/row
     dmat = new std::shared_ptr<xgboost::DMatrix>(std::move(sp_dmat));
     num_cols = 5;
  }

  // parameters for finding quantiles
  const std::string max_bin = "20";
  const std::string debug_synchronize = "true";
#define CONVERT_TO_STR(VAR, VAL) \
  { \
    std::stringstream sstr; \
    sstr << VAL; \
    VAR = sstr.str(); \
  }
  std::string n_gpus; CONVERT_TO_STR(n_gpus, (devices.Size()))
  // Training every row in a single GPU batch
  std::string gpu_batch_nrows; CONVERT_TO_STR(gpu_batch_nrows, -1)

  std::vector<std::pair<std::string, std::string>> training_params = {
    {"max_bin", max_bin},
    {"debug_synchronize", debug_synchronize},
    {"n_gpus", n_gpus},
    {"gpu_batch_nrows", gpu_batch_nrows}
  };

  // find quantiles on the CPU
  HistCutMatrix hmat_cpu;
  hmat_cpu.Init((*dmat).get(), atoi(max_bin.c_str()));

  // find the cuts on the GPU
  tree::GPUHistMakerSpecialised<GradientPairPrecise> hist_maker;
  hist_maker.Init(training_params);
  hist_maker.InitDataOnce(dmat->get());
  const HistCutMatrix &hmat_gpu = hist_maker.hmat_;

  // compare the cuts
  double eps = 1e-2;
  ASSERT_EQ(hmat_gpu.min_val.size(), num_cols);
  ASSERT_EQ(hmat_gpu.row_ptr.size(), num_cols + 1);
  ASSERT_EQ(hmat_gpu.cut.size(), hmat_cpu.cut.size());
  ASSERT_LT(fabs(hmat_cpu.min_val[0] - hmat_gpu.min_val[0]), eps * nrows);
  for (int i = 0; i < hmat_gpu.cut.size(); ++i) {
    ASSERT_LT(fabs(hmat_cpu.cut[i] - hmat_gpu.cut[i]), eps * nrows);
  }

  delete dmat;
}

TEST(gpu_hist_util, DeviceSketch) {
  TestDeviceSketch(GPUSet::Range(0, 1));
}

TEST(gpu_hist_util, DeviceSketch_ExternalMemory) {
  TestDeviceSketch(GPUSet::Range(0, 1), true);
}

#if defined(XGBOOST_USE_NCCL)
TEST(gpu_hist_util, MGPU_DeviceSketch) {
  auto devices = GPUSet::AllVisible();
  CHECK_GT(devices.Size(), 1);
  TestDeviceSketch(devices);
}

TEST(gpu_hist_util, MGPU_DeviceSketch_ExternalMemory) {
  auto devices = GPUSet::AllVisible();
  CHECK_GT(devices.Size(), 1);
  TestDeviceSketch(devices, true);
}
#endif

}  // namespace common
}  // namespace xgboost
