#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>

#include "gtest/gtest.h"

#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>

#include "xgboost/c_api.h"

#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/hist_util.h"

#include "../helpers.h"

namespace xgboost {
namespace common {

void TestDeviceSketch(const GPUSet& devices, bool use_external_memory) {
  // create the data
  int nrows = 10001;
  std::shared_ptr<xgboost::DMatrix> *dmat = nullptr;

  size_t num_cols = 1;
  if (use_external_memory) {
     auto sp_dmat = CreateSparsePageDMatrix(nrows * 3, 128UL); // 3 entries/row
     dmat = new std::shared_ptr<xgboost::DMatrix>(std::move(sp_dmat));
     num_cols = 5;
  } else {
     std::vector<float> test_data(nrows);
     auto count_iter = thrust::make_counting_iterator(0);
     // fill in reverse order
     std::copy(count_iter, count_iter + nrows, test_data.rbegin());

     // create the DMatrix
     DMatrixHandle dmat_handle;
     XGDMatrixCreateFromMat(test_data.data(), nrows, 1, -1,
                            &dmat_handle);
     dmat = static_cast<std::shared_ptr<xgboost::DMatrix> *>(dmat_handle);
  }

  tree::TrainParam p;
  p.max_bin = 20;
  int gpu_batch_nrows = 0;

  // find quantiles on the CPU
  HistCutMatrix hmat_cpu;
  hmat_cpu.Init((*dmat).get(), p.max_bin);

  // find the cuts on the GPU
  HistCutMatrix hmat_gpu;
  size_t row_stride = DeviceSketch(p, CreateEmptyGenericParam(0, devices.Size()), gpu_batch_nrows,
                                   dmat->get(), &hmat_gpu);

  // compare the row stride with the one obtained from the dmatrix
  size_t expected_row_stride = 0;
  for (const auto &batch : dmat->get()->GetRowBatches()) {
    const auto &offset_vec = batch.offset.ConstHostVector();
    for (int i = 1; i <= offset_vec.size() -1; ++i) {
      expected_row_stride = std::max(expected_row_stride, offset_vec[i] - offset_vec[i-1]);
    }
  }

  ASSERT_EQ(expected_row_stride, row_stride);

  // compare the cuts
  double eps = 1e-2;
  ASSERT_EQ(hmat_gpu.min_val.size(), num_cols);
  ASSERT_EQ(hmat_gpu.row_ptr.size(), num_cols + 1);
  ASSERT_EQ(hmat_gpu.cut.size(), hmat_cpu.cut.size());
  ASSERT_LT(fabs(hmat_cpu.min_val[0] - hmat_gpu.min_val[0]), eps * nrows);
  for (int i = 0; i < hmat_gpu.cut.size(); ++i) {
    ASSERT_LT(fabs(hmat_cpu.cut[i] - hmat_gpu.cut[i]), eps * nrows);
  }

  delete dmat;
}

TEST(gpu_hist_util, DeviceSketch) {
  TestDeviceSketch(GPUSet::Range(0, 1), false);
}

TEST(gpu_hist_util, DeviceSketch_ExternalMemory) {
  TestDeviceSketch(GPUSet::Range(0, 1), true);
}

#if defined(XGBOOST_USE_NCCL)
TEST(gpu_hist_util, MGPU_DeviceSketch) {
  auto devices = GPUSet::AllVisible();
  CHECK_GT(devices.Size(), 1);
  TestDeviceSketch(devices, false);
}

TEST(gpu_hist_util, MGPU_DeviceSketch_ExternalMemory) {
  auto devices = GPUSet::AllVisible();
  CHECK_GT(devices.Size(), 1);
  TestDeviceSketch(devices, true);
}
#endif

}  // namespace common
}  // namespace xgboost
