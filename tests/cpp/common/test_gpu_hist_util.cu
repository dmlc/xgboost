#include "hip/hip_runtime.h"
#include <dmlc/filesystem.h>
#include <gtest/gtest.h>

#include <algorithm>
#include <cmath>


#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>

#include "xgboost/c_api.h"

#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/hist_util.h"

#include "../helpers.h"

namespace xgboost {
namespace common {

void TestDeviceSketch(bool use_external_memory) {
  // create the data
  int nrows = 10001;
  std::shared_ptr<xgboost::DMatrix> *dmat = nullptr;

  size_t num_cols = 1;
  dmlc::TemporaryDirectory tmpdir;
  std::string file = tmpdir.path + "/big.libsvm";
  if (use_external_memory) {
    auto sp_dmat = CreateSparsePageDMatrix(nrows * 3, 128UL, file); // 3 entries/row
    dmat = new std::shared_ptr<xgboost::DMatrix>(std::move(sp_dmat));
    num_cols = 5;
  } else {
     std::vector<float> test_data(nrows);
     auto count_iter = thrust::make_counting_iterator(0);
     // fill in reverse order
     std::copy(count_iter, count_iter + nrows, test_data.rbegin());

     // create the DMatrix
     DMatrixHandle dmat_handle;
     XGDMatrixCreateFromMat(test_data.data(), nrows, 1, -1,
                            &dmat_handle);
     dmat = static_cast<std::shared_ptr<xgboost::DMatrix> *>(dmat_handle);
  }

  int device{0};
  int max_bin{20};
  int gpu_batch_nrows{0};

  // find quantiles on the CPU
  HistogramCuts hmat_cpu;
  hmat_cpu.Build((*dmat).get(), max_bin);

  // find the cuts on the GPU
  HistogramCuts hmat_gpu;
  size_t row_stride = DeviceSketch(device, max_bin, gpu_batch_nrows, dmat->get(), &hmat_gpu);

  // compare the row stride with the one obtained from the dmatrix
  bst_row_t expected_row_stride = 0;
  for (const auto &batch : dmat->get()->GetBatches<xgboost::SparsePage>()) {
    const auto &offset_vec = batch.offset.ConstHostVector();
    for (int i = 1; i <= offset_vec.size() -1; ++i) {
      expected_row_stride = std::max(expected_row_stride, offset_vec[i] - offset_vec[i-1]);
    }
  }

  ASSERT_EQ(expected_row_stride, row_stride);

  // compare the cuts
  double eps = 1e-2;
  ASSERT_EQ(hmat_gpu.MinValues().size(), num_cols);
  ASSERT_EQ(hmat_gpu.Ptrs().size(), num_cols + 1);
  ASSERT_EQ(hmat_gpu.Values().size(), hmat_cpu.Values().size());
  ASSERT_LT(fabs(hmat_cpu.MinValues()[0] - hmat_gpu.MinValues()[0]), eps * nrows);
  for (int i = 0; i < hmat_gpu.Values().size(); ++i) {
    ASSERT_LT(fabs(hmat_cpu.Values()[i] - hmat_gpu.Values()[i]), eps * nrows);
  }

  delete dmat;
}

TEST(gpu_hist_util, DeviceSketch) {
  TestDeviceSketch(false);
}

TEST(gpu_hist_util, DeviceSketch_ExternalMemory) {
  TestDeviceSketch(true);
}

}  // namespace common
}  // namespace xgboost
