#include "hip/hip_runtime.h"
#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/hist_util.h"
#include "gtest/gtest.h"
#include "xgboost/c_api.h"
#include <algorithm>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>

namespace xgboost {
namespace common {

void TestDeviceSketch(const GPUSet& devices) {
  // create the data
  int nrows = 10001;
  std::vector<float> test_data(nrows);
  auto count_iter = thrust::make_counting_iterator(0);
  // fill in reverse order
  std::copy(count_iter, count_iter + nrows, test_data.rbegin());

  // create the DMatrix
  DMatrixHandle dmat_handle;
  XGDMatrixCreateFromMat(test_data.data(), nrows, 1, -1,
                         &dmat_handle);
  auto dmat = static_cast<std::shared_ptr<xgboost::DMatrix> *>(dmat_handle);

  // parameters for finding quantiles
  tree::TrainParam p;
  p.max_bin = 20;
  p.gpu_id = 0;
  p.n_gpus = devices.Size();
  // ensure that the exact quantiles are found
  p.gpu_batch_nrows = nrows * 10;

  // find quantiles on the CPU
  HistCutMatrix hmat_cpu;
  hmat_cpu.Init((*dmat).get(), p.max_bin);

  // find the cuts on the GPU
  const SparsePage& batch = *(*dmat)->GetRowBatches().begin();
  HistCutMatrix hmat_gpu;
  DeviceSketch(batch, (*dmat)->Info(), p, &hmat_gpu);

  // compare the cuts
  double eps = 1e-2;
  ASSERT_EQ(hmat_gpu.min_val.size(), 1);
  ASSERT_EQ(hmat_gpu.row_ptr.size(), 2);
  ASSERT_EQ(hmat_gpu.cut.size(), hmat_cpu.cut.size());
  ASSERT_LT(fabs(hmat_cpu.min_val[0] - hmat_gpu.min_val[0]), eps * nrows);
  for (int i = 0; i < hmat_gpu.cut.size(); ++i) {
    ASSERT_LT(fabs(hmat_cpu.cut[i] - hmat_gpu.cut[i]), eps * nrows);
  }

  delete dmat;
}

TEST(gpu_hist_util, DeviceSketch) {
  TestDeviceSketch(GPUSet::Range(0, 1));
}

#if defined(XGBOOST_USE_NCCL)
TEST(gpu_hist_util, MGPU_DeviceSketch) {
  auto devices = GPUSet::AllVisible();
  CHECK_GT(devices.Size(), 1);
  TestDeviceSketch(devices);
}
#endif

}  // namespace common
}  // namespace xgboost
