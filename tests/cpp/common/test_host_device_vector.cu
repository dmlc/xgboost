/*!
 * Copyright 2018 XGBoost contributors
 */

#include <gtest/gtest.h>
#include <thrust/equal.h>
#include <thrust/iterator/counting_iterator.h>

#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/host_device_vector.h"

namespace xgboost {
namespace common {

void SetDevice(int device) {
  int n_devices;
  dh::safe_cuda(hipGetDeviceCount(&n_devices));
  device %= n_devices;
  dh::safe_cuda(hipSetDevice(device));
}

struct HostDeviceVectorSetDeviceHandler {
  template <typename Functor>
  explicit HostDeviceVectorSetDeviceHandler(Functor f) {
    SetCudaSetDeviceHandler(f);
  }

  ~HostDeviceVectorSetDeviceHandler() {
    SetCudaSetDeviceHandler(nullptr);
  }
};

void InitHostDeviceVector(size_t n, const GPUDistribution& distribution,
                     HostDeviceVector<int> *v) {
  // create the vector
  GPUSet devices = distribution.Devices();
  v->Shard(distribution);
  v->Resize(n);

  ASSERT_EQ(v->Size(), n);
  ASSERT_TRUE(v->Distribution() == distribution);
  ASSERT_TRUE(v->Devices() == devices);
  // ensure that the devices have read-write access
  for (int i = 0; i < devices.Size(); ++i) {
    ASSERT_TRUE(v->DeviceCanAccess(i, GPUAccess::kRead));
    ASSERT_TRUE(v->DeviceCanAccess(i, GPUAccess::kWrite));
  }
  // ensure that the host has no access
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kWrite));
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kRead));

  // fill in the data on the host
  std::vector<int>& data_h = v->HostVector();
  // ensure that the host has full access, while the devices have none
  ASSERT_TRUE(v->HostCanAccess(GPUAccess::kRead));
  ASSERT_TRUE(v->HostCanAccess(GPUAccess::kWrite));
  for (int i = 0; i < devices.Size(); ++i) {
    ASSERT_FALSE(v->DeviceCanAccess(i, GPUAccess::kRead));
    ASSERT_FALSE(v->DeviceCanAccess(i, GPUAccess::kWrite));
  }
  ASSERT_EQ(data_h.size(), n);
  std::copy_n(thrust::make_counting_iterator(0), n, data_h.begin());
}

void PlusOne(HostDeviceVector<int> *v) {
  int n_devices = v->Devices().Size();
  for (int i = 0; i < n_devices; ++i) {
    SetDevice(i);
    thrust::transform(v->tbegin(i), v->tend(i), v->tbegin(i),
                      [=]__device__(unsigned int a){ return a + 1; });
  }
}

void CheckDevice(HostDeviceVector<int> *v,
                 const std::vector<size_t>& starts,
                 const std::vector<size_t>& sizes,
                 unsigned int first, GPUAccess access) {
  int n_devices = sizes.size();
  ASSERT_EQ(v->Devices().Size(), n_devices);
  for (int i = 0; i < n_devices; ++i) {
    ASSERT_EQ(v->DeviceSize(i), sizes.at(i));
    SetDevice(i);
    ASSERT_TRUE(thrust::equal(v->tcbegin(i), v->tcend(i),
                              thrust::make_counting_iterator(first + starts[i])));
    ASSERT_TRUE(v->DeviceCanAccess(i, GPUAccess::kRead));
    // ensure that the device has at most the access specified by access
    ASSERT_EQ(v->DeviceCanAccess(i, GPUAccess::kWrite), access == GPUAccess::kWrite);
  }
  ASSERT_EQ(v->HostCanAccess(GPUAccess::kRead), access == GPUAccess::kRead);
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kWrite));
  for (int i = 0; i < n_devices; ++i) {
    SetDevice(i);
    ASSERT_TRUE(thrust::equal(v->tbegin(i), v->tend(i),
                              thrust::make_counting_iterator(first + starts[i])));
    ASSERT_TRUE(v->DeviceCanAccess(i, GPUAccess::kRead));
    ASSERT_TRUE(v->DeviceCanAccess(i, GPUAccess::kWrite));
  }
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kRead));
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kWrite));
}

void CheckHost(HostDeviceVector<int> *v, GPUAccess access) {
  const std::vector<int>& data_h = access == GPUAccess::kWrite ?
    v->HostVector() : v->ConstHostVector();
  for (size_t i = 0; i < v->Size(); ++i) {
    ASSERT_EQ(data_h.at(i), i + 1);
  }
  ASSERT_TRUE(v->HostCanAccess(GPUAccess::kRead));
  ASSERT_EQ(v->HostCanAccess(GPUAccess::kWrite), access == GPUAccess::kWrite);
  size_t n_devices = v->Devices().Size();
  for (int i = 0; i < n_devices; ++i) {
    ASSERT_EQ(v->DeviceCanAccess(i, GPUAccess::kRead), access == GPUAccess::kRead);
    // the devices should have no write access
    ASSERT_FALSE(v->DeviceCanAccess(i, GPUAccess::kWrite));
  }
}

void TestHostDeviceVector
(size_t n, const GPUDistribution& distribution,
 const std::vector<size_t>& starts, const std::vector<size_t>& sizes) {
  HostDeviceVectorSetDeviceHandler hdvec_dev_hndlr(SetDevice);
  HostDeviceVector<int> v;
  InitHostDeviceVector(n, distribution, &v);
  CheckDevice(&v, starts, sizes, 0, GPUAccess::kRead);
  PlusOne(&v);
  CheckDevice(&v, starts, sizes, 1, GPUAccess::kWrite);
  CheckHost(&v, GPUAccess::kRead);
  CheckHost(&v, GPUAccess::kWrite);
}

TEST(HostDeviceVector, TestBlock) {
  size_t n = 1001;
  int n_devices = 2;
  auto distribution = GPUDistribution::Block(GPUSet::Range(0, n_devices));
  std::vector<size_t> starts{0, 501};
  std::vector<size_t> sizes{501, 500};
  TestHostDeviceVector(n, distribution, starts, sizes);
}

TEST(HostDeviceVector, TestGranular) {
  size_t n = 3003;
  int n_devices = 2;
  auto distribution = GPUDistribution::Granular(GPUSet::Range(0, n_devices), 3);
  std::vector<size_t> starts{0, 1503};
  std::vector<size_t> sizes{1503, 1500};
  TestHostDeviceVector(n, distribution, starts, sizes);
}

TEST(HostDeviceVector, TestOverlap) {
  size_t n = 1001;
  int n_devices = 2;
  auto distribution = GPUDistribution::Overlap(GPUSet::Range(0, n_devices), 1);
  std::vector<size_t> starts{0, 500};
  std::vector<size_t> sizes{501, 501};
  TestHostDeviceVector(n, distribution, starts, sizes);
}

TEST(HostDeviceVector, TestExplicit) {
  size_t n = 1001;
  int n_devices = 2;
  std::vector<size_t> offsets{0, 550, 1001};
  auto distribution = GPUDistribution::Explicit(GPUSet::Range(0, n_devices), offsets);
  std::vector<size_t> starts{0, 550};
  std::vector<size_t> sizes{550, 451};
  TestHostDeviceVector(n, distribution, starts, sizes);
}

TEST(HostDeviceVector, TestCopy) {
  size_t n = 1001;
  int n_devices = 2;
  auto distribution = GPUDistribution::Block(GPUSet::Range(0, n_devices));
  std::vector<size_t> starts{0, 501};
  std::vector<size_t> sizes{501, 500};
  HostDeviceVectorSetDeviceHandler hdvec_dev_hndlr(SetDevice);

  HostDeviceVector<int> v;
  {
    // a separate scope to ensure that v1 is gone before further checks
    HostDeviceVector<int> v1;
    InitHostDeviceVector(n, distribution, &v1);
    v = v1;
  }
  CheckDevice(&v, starts, sizes, 0, GPUAccess::kRead);
  PlusOne(&v);
  CheckDevice(&v, starts, sizes, 1, GPUAccess::kWrite);
  CheckHost(&v, GPUAccess::kRead);
  CheckHost(&v, GPUAccess::kWrite);
}

TEST(HostDeviceVector, Shard) {
  std::vector<int> h_vec (2345);
  for (size_t i = 0; i < h_vec.size(); ++i) {
    h_vec[i] = i;
  }
  HostDeviceVector<int> vec (h_vec);
  auto devices = GPUSet::Range(0, 1);

  vec.Shard(devices);
  ASSERT_EQ(vec.DeviceSize(0), h_vec.size());
  ASSERT_EQ(vec.Size(), h_vec.size());
  auto span = vec.DeviceSpan(0);  // sync to device

  vec.Reshard(GPUDistribution::Empty());  // pull back to cpu, empty devices.
  ASSERT_EQ(vec.Size(), h_vec.size());
  ASSERT_TRUE(vec.Devices().IsEmpty());

  auto h_vec_1 = vec.HostVector();
  ASSERT_TRUE(std::equal(h_vec_1.cbegin(), h_vec_1.cend(), h_vec.cbegin()));
}

TEST(HostDeviceVector, Reshard) {
  std::vector<int> h_vec (2345);
  for (size_t i = 0; i < h_vec.size(); ++i) {
    h_vec[i] = i;
  }
  HostDeviceVector<int> vec (h_vec);
  auto devices = GPUSet::Range(0, 1);

  vec.Shard(devices);
  ASSERT_EQ(vec.DeviceSize(0), h_vec.size());
  ASSERT_EQ(vec.Size(), h_vec.size());
  PlusOne(&vec);

  vec.Reshard(GPUDistribution::Empty());
  ASSERT_EQ(vec.Size(), h_vec.size());
  ASSERT_TRUE(vec.Devices().IsEmpty());

  auto h_vec_1 = vec.HostVector();
  for (size_t i = 0; i < h_vec_1.size(); ++i) {
    ASSERT_EQ(h_vec_1.at(i), i + 1);
  }
}

TEST(HostDeviceVector, Span) {
  HostDeviceVector<float> vec {1.0f, 2.0f, 3.0f, 4.0f};
  vec.Shard(GPUSet{0, 1});
  auto span = vec.DeviceSpan(0);
  ASSERT_EQ(vec.DeviceSize(0), span.size());
  ASSERT_EQ(vec.DevicePointer(0), span.data());
  auto const_span = vec.ConstDeviceSpan(0);
  ASSERT_EQ(vec.DeviceSize(0), span.size());
  ASSERT_EQ(vec.ConstDevicePointer(0), span.data());
}

// Multi-GPUs' test
#if defined(XGBOOST_USE_NCCL)
TEST(HostDeviceVector, MGPU_Shard) {
  auto devices = GPUSet::AllVisible();
  if (devices.Size() < 2) {
    LOG(WARNING) << "Not testing in multi-gpu environment.";
    return;
  }

  std::vector<int> h_vec (2345);
  for (size_t i = 0; i < h_vec.size(); ++i) {
    h_vec[i] = i;
  }
  HostDeviceVector<int> vec (h_vec);

  // Data size for each device.
  std::vector<size_t> devices_size (devices.Size());

  // From CPU to GPUs.
  vec.Shard(devices);
  size_t total_size = 0;
  for (size_t i = 0; i < devices.Size(); ++i) {
    total_size += vec.DeviceSize(i);
    devices_size[i] = vec.DeviceSize(i);
  }
  ASSERT_EQ(total_size, h_vec.size());
  ASSERT_EQ(total_size, vec.Size());

  // Shard from devices to devices with different distribution.
  EXPECT_ANY_THROW(
      vec.Shard(GPUDistribution::Granular(devices, 12)));

  // All data is drawn back to CPU
  vec.Reshard(GPUDistribution::Empty());
  ASSERT_TRUE(vec.Devices().IsEmpty());
  ASSERT_EQ(vec.Size(), h_vec.size());

  vec.Shard(GPUDistribution::Granular(devices, 12));
  total_size = 0;
  for (size_t i = 0; i < devices.Size(); ++i) {
    total_size += vec.DeviceSize(i);
    devices_size[i] = vec.DeviceSize(i);
  }
  ASSERT_EQ(total_size, h_vec.size());
  ASSERT_EQ(total_size, vec.Size());
}

TEST(HostDeviceVector, MGPU_Reshard) {
  auto devices = GPUSet::AllVisible();
  if (devices.Size() < 2) {
    LOG(WARNING) << "Not testing in multi-gpu environment.";
    return;
  }

  size_t n = 1001;
  int n_devices = 2;
  auto distribution = GPUDistribution::Block(GPUSet::Range(0, n_devices));
  std::vector<size_t> starts{0, 501};
  std::vector<size_t> sizes{501, 500};

  HostDeviceVector<int> v;
  InitHostDeviceVector(n, distribution, &v);
  CheckDevice(&v, starts, sizes, 0, GPUAccess::kRead);
  PlusOne(&v);
  CheckDevice(&v, starts, sizes, 1, GPUAccess::kWrite);
  CheckHost(&v, GPUAccess::kRead);
  CheckHost(&v, GPUAccess::kWrite);

  auto distribution1 = GPUDistribution::Overlap(GPUSet::Range(0, n_devices), 1);
  v.Reshard(distribution1);

  for (size_t i = 0; i < n_devices; ++i) {
    auto span = v.DeviceSpan(i);  // sync to device
  }

  std::vector<size_t> starts1{0, 500};
  std::vector<size_t> sizes1{501, 501};
  CheckDevice(&v, starts1, sizes1, 1, GPUAccess::kWrite);
  CheckHost(&v, GPUAccess::kRead);
  CheckHost(&v, GPUAccess::kWrite);
}
#endif

}  // namespace common
}  // namespace xgboost
