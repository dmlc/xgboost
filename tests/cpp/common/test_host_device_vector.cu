/*!
 * Copyright 2018 XGBoost contributors
 */

#include <gtest/gtest.h>
#include <thrust/equal.h>
#include <thrust/iterator/counting_iterator.h>

#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/host_device_vector.h"

namespace xgboost {
namespace common {

void SetDevice(int device) {
  int n_devices;
  dh::safe_cuda(hipGetDeviceCount(&n_devices));
  device %= n_devices;
  dh::safe_cuda(hipSetDevice(device));
}

void InitHostDeviceVector(size_t n, const GPUDistribution& distribution,
                     HostDeviceVector<int> *v) {
  // create the vector
  GPUSet devices = distribution.Devices();
  v->Reshard(distribution);
  v->Resize(n);

  ASSERT_EQ(v->Size(), n);
  ASSERT_TRUE(v->Distribution() == distribution);
  ASSERT_TRUE(v->Devices() == devices);
  // ensure that the devices have read-write access
  for (int i = 0; i < devices.Size(); ++i) {
    ASSERT_TRUE(v->DeviceCanAccess(i, GPUAccess::kRead));
    ASSERT_TRUE(v->DeviceCanAccess(i, GPUAccess::kWrite));
  }
  // ensure that the host has no access
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kWrite));
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kRead));

  // fill in the data on the host
  std::vector<int>& data_h = v->HostVector();
  // ensure that the host has full access, while the devices have none
  ASSERT_TRUE(v->HostCanAccess(GPUAccess::kRead));
  ASSERT_TRUE(v->HostCanAccess(GPUAccess::kWrite));
  for (int i = 0; i < devices.Size(); ++i) {
    ASSERT_FALSE(v->DeviceCanAccess(i, GPUAccess::kRead));
    ASSERT_FALSE(v->DeviceCanAccess(i, GPUAccess::kWrite));
  }
  ASSERT_EQ(data_h.size(), n);
  std::copy_n(thrust::make_counting_iterator(0), n, data_h.begin());
}

void PlusOne(HostDeviceVector<int> *v) {
  int n_devices = v->Devices().Size();
  for (int i = 0; i < n_devices; ++i) {
    SetDevice(i);
    thrust::transform(v->tbegin(i), v->tend(i), v->tbegin(i),
                      [=]__device__(unsigned int a){ return a + 1; });
  }
}

void CheckDevice(HostDeviceVector<int> *v,
                 const std::vector<size_t>& starts,
                 const std::vector<size_t>& sizes,
                 unsigned int first, GPUAccess access) {
  int n_devices = sizes.size();
  ASSERT_EQ(v->Devices().Size(), n_devices);
  for (int i = 0; i < n_devices; ++i) {
    ASSERT_EQ(v->DeviceSize(i), sizes.at(i));
    SetDevice(i);
    ASSERT_TRUE(thrust::equal(v->tcbegin(i), v->tcend(i),
                              thrust::make_counting_iterator(first + starts[i])));
    ASSERT_TRUE(v->DeviceCanAccess(i, GPUAccess::kRead));
    // ensure that the device has at most the access specified by access
    ASSERT_EQ(v->DeviceCanAccess(i, GPUAccess::kWrite), access == GPUAccess::kWrite);
  }
  ASSERT_EQ(v->HostCanAccess(GPUAccess::kRead), access == GPUAccess::kRead);
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kWrite));
  for (int i = 0; i < n_devices; ++i) {
    SetDevice(i);
    ASSERT_TRUE(thrust::equal(v->tbegin(i), v->tend(i),
                              thrust::make_counting_iterator(first + starts[i])));
    ASSERT_TRUE(v->DeviceCanAccess(i, GPUAccess::kRead));
    ASSERT_TRUE(v->DeviceCanAccess(i, GPUAccess::kWrite));
  }
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kRead));
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kWrite));
}

void CheckHost(HostDeviceVector<int> *v, GPUAccess access) {
  const std::vector<int>& data_h = access == GPUAccess::kWrite ?
    v->HostVector() : v->ConstHostVector();
  for (size_t i = 0; i < v->Size(); ++i) {
    ASSERT_EQ(data_h.at(i), i + 1);
  }
  ASSERT_TRUE(v->HostCanAccess(GPUAccess::kRead));
  ASSERT_EQ(v->HostCanAccess(GPUAccess::kWrite), access == GPUAccess::kWrite);
  size_t n_devices = v->Devices().Size();
  for (int i = 0; i < n_devices; ++i) {
    ASSERT_EQ(v->DeviceCanAccess(i, GPUAccess::kRead), access == GPUAccess::kRead);
    // the devices should have no write access
    ASSERT_FALSE(v->DeviceCanAccess(i, GPUAccess::kWrite));
  }
}

void TestHostDeviceVector
(size_t n, const GPUDistribution& distribution,
 const std::vector<size_t>& starts, const std::vector<size_t>& sizes) {
  SetCudaSetDeviceHandler(SetDevice);
  HostDeviceVector<int> v;
  InitHostDeviceVector(n, distribution, &v);
  CheckDevice(&v, starts, sizes, 0, GPUAccess::kRead);
  PlusOne(&v);
  CheckDevice(&v, starts, sizes, 1, GPUAccess::kWrite);
  CheckHost(&v, GPUAccess::kRead);
  CheckHost(&v, GPUAccess::kWrite);
  SetCudaSetDeviceHandler(nullptr);
}

TEST(HostDeviceVector, TestBlock) {
  size_t n = 1001;
  int n_devices = 2;
  auto distribution = GPUDistribution::Block(GPUSet::Range(0, n_devices));
  std::vector<size_t> starts{0, 501};
  std::vector<size_t> sizes{501, 500};
  TestHostDeviceVector(n, distribution, starts, sizes);
}

TEST(HostDeviceVector, TestGranular) {
  size_t n = 3003;
  int n_devices = 2;
  auto distribution = GPUDistribution::Granular(GPUSet::Range(0, n_devices), 3);
  std::vector<size_t> starts{0, 1503};
  std::vector<size_t> sizes{1503, 1500};
  TestHostDeviceVector(n, distribution, starts, sizes);
}

TEST(HostDeviceVector, TestOverlap) {
  size_t n = 1001;
  int n_devices = 2;
  auto distribution = GPUDistribution::Overlap(GPUSet::Range(0, n_devices), 1);
  std::vector<size_t> starts{0, 500};
  std::vector<size_t> sizes{501, 501};
  TestHostDeviceVector(n, distribution, starts, sizes);
}

TEST(HostDeviceVector, TestExplicit) {
  size_t n = 1001;
  int n_devices = 2;
  std::vector<size_t> offsets{0, 550, 1001};
  auto distribution = GPUDistribution::Explicit(GPUSet::Range(0, n_devices), offsets);
  std::vector<size_t> starts{0, 550};
  std::vector<size_t> sizes{550, 451};
  TestHostDeviceVector(n, distribution, starts, sizes);
}

TEST(HostDeviceVector, TestCopy) {
  size_t n = 1001;
  int n_devices = 2;
  auto distribution = GPUDistribution::Block(GPUSet::Range(0, n_devices));
  std::vector<size_t> starts{0, 501};
  std::vector<size_t> sizes{501, 500};
  SetCudaSetDeviceHandler(SetDevice);

  HostDeviceVector<int> v;
  {
    // a separate scope to ensure that v1 is gone before further checks
    HostDeviceVector<int> v1;
    InitHostDeviceVector(n, distribution, &v1);
    v = v1;
  }
  CheckDevice(&v, starts, sizes, 0, GPUAccess::kRead);
  PlusOne(&v);
  CheckDevice(&v, starts, sizes, 1, GPUAccess::kWrite);
  CheckHost(&v, GPUAccess::kRead);
  CheckHost(&v, GPUAccess::kWrite);
  SetCudaSetDeviceHandler(nullptr);
}

TEST(HostDeviceVector, Reshard) {
  std::vector<int> h_vec (2345);
  for (size_t i = 0; i < h_vec.size(); ++i) {
    h_vec[i] = i;
  }
  HostDeviceVector<int> vec (h_vec);
  auto devices = GPUSet::Range(0, 1);

  vec.Reshard(devices);
  ASSERT_EQ(vec.DeviceSize(0), h_vec.size());
  ASSERT_EQ(vec.Size(), h_vec.size());
  auto span = vec.DeviceSpan(0);  // sync to device

  vec.Reshard(GPUSet::Empty());  // pull back to cpu, empty devices.
  ASSERT_EQ(vec.Size(), h_vec.size());
  ASSERT_TRUE(vec.Devices().IsEmpty());

  auto h_vec_1 = vec.HostVector();
  ASSERT_TRUE(std::equal(h_vec_1.cbegin(), h_vec_1.cend(), h_vec.cbegin()));
}

TEST(HostDeviceVector, Span) {
  HostDeviceVector<float> vec {1.0f, 2.0f, 3.0f, 4.0f};
  vec.Reshard(GPUSet{0, 1});
  auto span = vec.DeviceSpan(0);
  ASSERT_EQ(vec.DeviceSize(0), span.size());
  ASSERT_EQ(vec.DevicePointer(0), span.data());
  auto const_span = vec.ConstDeviceSpan(0);
  ASSERT_EQ(vec.DeviceSize(0), span.size());
  ASSERT_EQ(vec.ConstDevicePointer(0), span.data());
}

// Multi-GPUs' test
#if defined(XGBOOST_USE_NCCL)
TEST(HostDeviceVector, MGPU_Reshard) {
  auto devices = GPUSet::AllVisible();
  if (devices.Size() < 2) {
    LOG(WARNING) << "Not testing in multi-gpu environment.";
    return;
  }

  std::vector<int> h_vec (2345);
  for (size_t i = 0; i < h_vec.size(); ++i) {
    h_vec[i] = i;
  }
  HostDeviceVector<int> vec (h_vec);

  // Data size for each device.
  std::vector<size_t> devices_size (devices.Size());

  // From CPU to GPUs.
  vec.Reshard(devices);
  size_t total_size = 0;
  for (size_t i = 0; i < devices.Size(); ++i) {
    total_size += vec.DeviceSize(i);
    devices_size[i] = vec.DeviceSize(i);
  }
  ASSERT_EQ(total_size, h_vec.size());
  ASSERT_EQ(total_size, vec.Size());

  // Reshard from devices to devices with different distribution.
  EXPECT_ANY_THROW(
      vec.Reshard(GPUDistribution::Granular(devices, 12)));

  // All data is drawn back to CPU
  vec.Reshard(GPUSet::Empty());
  ASSERT_TRUE(vec.Devices().IsEmpty());
  ASSERT_EQ(vec.Size(), h_vec.size());

  vec.Reshard(GPUDistribution::Granular(devices, 12));
  total_size = 0;
  for (size_t i = 0; i < devices.Size(); ++i) {
    total_size += vec.DeviceSize(i);
    devices_size[i] = vec.DeviceSize(i);
  }
  ASSERT_EQ(total_size, h_vec.size());
  ASSERT_EQ(total_size, vec.Size());
}
#endif

}  // namespace common
}  // namespace xgboost
