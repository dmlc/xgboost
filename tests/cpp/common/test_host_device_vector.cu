/*!
 * Copyright 2018 XGBoost contributors
 */

#include <gtest/gtest.h>
#include <thrust/equal.h>
#include <thrust/iterator/counting_iterator.h>

#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/host_device_vector.h"

namespace xgboost {
namespace common {

void SetDevice(int device) {
  int n_devices;
  dh::safe_cuda(hipGetDeviceCount(&n_devices));
  device %= n_devices;
  dh::safe_cuda(hipSetDevice(device));
}

struct HostDeviceVectorSetDeviceHandler {
  template <typename Functor>
  explicit HostDeviceVectorSetDeviceHandler(Functor f) {
    SetCudaSetDeviceHandler(f);
  }

  ~HostDeviceVectorSetDeviceHandler() {
    SetCudaSetDeviceHandler(nullptr);
  }
};

void InitHostDeviceVector(size_t n, int device, HostDeviceVector<int> *v) {
  // create the vector
  v->SetDevice(device);
  v->Resize(n);

  ASSERT_EQ(v->Size(), n);
  ASSERT_EQ(v->DeviceIdx(), device);
  // ensure that the device have read-write access
  ASSERT_TRUE(v->DeviceCanAccess(GPUAccess::kRead));
  ASSERT_TRUE(v->DeviceCanAccess(GPUAccess::kWrite));
  // ensure that the host has no access
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kWrite));
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kRead));

  // fill in the data on the host
  std::vector<int>& data_h = v->HostVector();
  // ensure that the host has full access, while the device have none
  ASSERT_TRUE(v->HostCanAccess(GPUAccess::kRead));
  ASSERT_TRUE(v->HostCanAccess(GPUAccess::kWrite));
  ASSERT_FALSE(v->DeviceCanAccess(GPUAccess::kRead));
  ASSERT_FALSE(v->DeviceCanAccess(GPUAccess::kWrite));
  ASSERT_EQ(data_h.size(), n);
  std::copy_n(thrust::make_counting_iterator(0), n, data_h.begin());
}

void PlusOne(HostDeviceVector<int> *v) {
  int device = v->DeviceIdx();
  SetDevice(device);
  thrust::transform(v->tbegin(), v->tend(), v->tbegin(),
                    [=]__device__(unsigned int a){ return a + 1; });
}

void CheckDevice(HostDeviceVector<int> *v,
                 const std::vector<size_t>& starts,
                 const std::vector<size_t>& sizes,
                 unsigned int first, GPUAccess access) {
  int n_devices = sizes.size();
  ASSERT_EQ(n_devices, 1);
  for (int i = 0; i < n_devices; ++i) {
    ASSERT_EQ(v->DeviceSize(), sizes.at(i));
    SetDevice(i);
    ASSERT_TRUE(thrust::equal(v->tcbegin(), v->tcend(),
                              thrust::make_counting_iterator(first + starts[i])));
    ASSERT_TRUE(v->DeviceCanAccess(GPUAccess::kRead));
    // ensure that the device has at most the access specified by access
    ASSERT_EQ(v->DeviceCanAccess(GPUAccess::kWrite), access == GPUAccess::kWrite);
  }
  ASSERT_EQ(v->HostCanAccess(GPUAccess::kRead), access == GPUAccess::kRead);
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kWrite));
  for (int i = 0; i < n_devices; ++i) {
    SetDevice(i);
    ASSERT_TRUE(thrust::equal(v->tbegin(), v->tend(),
                              thrust::make_counting_iterator(first + starts[i])));
    ASSERT_TRUE(v->DeviceCanAccess(GPUAccess::kRead));
    ASSERT_TRUE(v->DeviceCanAccess(GPUAccess::kWrite));
  }
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kRead));
  ASSERT_FALSE(v->HostCanAccess(GPUAccess::kWrite));
}

void CheckHost(HostDeviceVector<int> *v, GPUAccess access) {
  const std::vector<int>& data_h = access == GPUAccess::kWrite ?
    v->HostVector() : v->ConstHostVector();
  for (size_t i = 0; i < v->Size(); ++i) {
    ASSERT_EQ(data_h.at(i), i + 1);
  }
  ASSERT_TRUE(v->HostCanAccess(GPUAccess::kRead));
  ASSERT_EQ(v->HostCanAccess(GPUAccess::kWrite), access == GPUAccess::kWrite);
  size_t n_devices = 1;
  for (int i = 0; i < n_devices; ++i) {
    ASSERT_EQ(v->DeviceCanAccess(GPUAccess::kRead), access == GPUAccess::kRead);
    // the devices should have no write access
    ASSERT_FALSE(v->DeviceCanAccess(GPUAccess::kWrite));
  }
}

void TestHostDeviceVector
(size_t n, int device,
 const std::vector<size_t>& starts, const std::vector<size_t>& sizes) {
  HostDeviceVectorSetDeviceHandler hdvec_dev_hndlr(SetDevice);
  HostDeviceVector<int> v;
  InitHostDeviceVector(n, device, &v);
  CheckDevice(&v, starts, sizes, 0, GPUAccess::kRead);
  PlusOne(&v);
  CheckDevice(&v, starts, sizes, 1, GPUAccess::kWrite);
  CheckHost(&v, GPUAccess::kRead);
  CheckHost(&v, GPUAccess::kWrite);
}

TEST(HostDeviceVector, TestBlock) {
  size_t n = 1001;
  int device = 0;
  std::vector<size_t> starts{0};
  std::vector<size_t> sizes{1001};
  TestHostDeviceVector(n, device, starts, sizes);
}

TEST(HostDeviceVector, TestCopy) {
  size_t n = 1001;
  int device = 0;
  std::vector<size_t> starts{0};
  std::vector<size_t> sizes{1001};
  HostDeviceVectorSetDeviceHandler hdvec_dev_hndlr(SetDevice);

  HostDeviceVector<int> v;
  {
    // a separate scope to ensure that v1 is gone before further checks
    HostDeviceVector<int> v1;
    InitHostDeviceVector(n, device, &v1);
    v = v1;
  }
  CheckDevice(&v, starts, sizes, 0, GPUAccess::kRead);
  PlusOne(&v);
  CheckDevice(&v, starts, sizes, 1, GPUAccess::kWrite);
  CheckHost(&v, GPUAccess::kRead);
  CheckHost(&v, GPUAccess::kWrite);
}

TEST(HostDeviceVector, Shard) {
  std::vector<int> h_vec (2345);
  for (size_t i = 0; i < h_vec.size(); ++i) {
    h_vec[i] = i;
  }
  HostDeviceVector<int> vec (h_vec);
  auto device = 0;

  vec.SetDevice(device);
  ASSERT_EQ(vec.DeviceSize(), h_vec.size());
  ASSERT_EQ(vec.Size(), h_vec.size());
  auto span = vec.DeviceSpan();  // sync to device

  vec.SetDevice(-1);  // pull back to cpu.
  ASSERT_EQ(vec.Size(), h_vec.size());
  ASSERT_EQ(vec.DeviceIdx(), -1);

  auto h_vec_1 = vec.HostVector();
  ASSERT_TRUE(std::equal(h_vec_1.cbegin(), h_vec_1.cend(), h_vec.cbegin()));
}

TEST(HostDeviceVector, Reshard) {
  std::vector<int> h_vec (2345);
  for (size_t i = 0; i < h_vec.size(); ++i) {
    h_vec[i] = i;
  }
  HostDeviceVector<int> vec (h_vec);
  auto device = 0;

  vec.SetDevice(device);
  ASSERT_EQ(vec.DeviceSize(), h_vec.size());
  ASSERT_EQ(vec.Size(), h_vec.size());
  PlusOne(&vec);

  vec.SetDevice(-1);
  ASSERT_EQ(vec.Size(), h_vec.size());
  ASSERT_EQ(vec.DeviceIdx(), -1);

  auto h_vec_1 = vec.HostVector();
  for (size_t i = 0; i < h_vec_1.size(); ++i) {
    ASSERT_EQ(h_vec_1.at(i), i + 1);
  }
}

TEST(HostDeviceVector, Span) {
  HostDeviceVector<float> vec {1.0f, 2.0f, 3.0f, 4.0f};
  vec.SetDevice(0);
  auto span = vec.DeviceSpan();
  ASSERT_EQ(vec.DeviceSize(), span.size());
  ASSERT_EQ(vec.DevicePointer(), span.data());
  auto const_span = vec.ConstDeviceSpan();
  ASSERT_EQ(vec.DeviceSize(), span.size());
  ASSERT_EQ(vec.ConstDevicePointer(), span.data());
}

}  // namespace common
}  // namespace xgboost
