#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include "test_quantile.h"
#include "../helpers.h"
#include "../../../src/common/hist_util.cuh"
#include "../../../src/common/quantile.cuh"

namespace xgboost {
namespace common {
TEST(GPUQuantile, Basic) {
  constexpr size_t kRows = 1000, kCols = 100, kBins = 256;
  SketchContainer sketch(kBins, kCols, kRows, 0);
  dh::caching_device_vector<SketchEntry> entries;
  dh::device_vector<bst_row_t> cuts_ptr(kCols+1);
  thrust::fill(cuts_ptr.begin(), cuts_ptr.end(), 0);
  // Push empty
  sketch.Push(dh::ToSpan(cuts_ptr), &entries);
  ASSERT_EQ(sketch.Data().size(), 0);
}

void TestSketchUnique(float sparsity) {
  constexpr size_t kRows = 1000, kCols = 100;
  RunWithSeedsAndBins(kRows, [kRows, kCols, sparsity](int32_t seed, size_t n_bins, MetaInfo const& info) {
    SketchContainer sketch(n_bins, kCols, kRows, 0);

    HostDeviceVector<float> storage;
    std::string interface_str = RandomDataGenerator{kRows, kCols, sparsity}
                                    .Seed(seed)
                                    .Device(0)
                                    .GenerateArrayInterface(&storage);
    data::CupyAdapter adapter(interface_str);
    AdapterDeviceSketch(adapter.Value(), n_bins, info,
                        std::numeric_limits<float>::quiet_NaN(), &sketch);
    auto n_cuts = detail::RequiredSampleCutsPerColumn(n_bins, kRows);

    dh::caching_device_vector<size_t> column_sizes_scan;
    HostDeviceVector<size_t> cut_sizes_scan;
    auto batch = adapter.Value();
    data::IsValidFunctor is_valid(std::numeric_limits<float>::quiet_NaN());
    auto batch_iter = dh::MakeTransformIterator<data::COOTuple>(
        thrust::make_counting_iterator(0llu),
        [=] __device__(size_t idx) { return batch.GetElement(idx); });
    auto end = kCols * kRows;
    detail::GetColumnSizesScan(0, kCols, n_cuts, batch_iter, is_valid, 0, end,
                               &cut_sizes_scan, &column_sizes_scan);
    auto const& cut_sizes = cut_sizes_scan.HostVector();

    if (sparsity == 0) {
      ASSERT_EQ(sketch.Data().size(), n_cuts * kCols);
    } else {
      ASSERT_EQ(sketch.Data().size(), cut_sizes.back());
    }

    sketch.Unique();
    ASSERT_TRUE(thrust::is_sorted(thrust::device, sketch.Data().data(),
                                  sketch.Data().data() + sketch.Data().size(),
                                  detail::SketchUnique{}));
  });
}

TEST(GPUQuantile, Unique) {
  TestSketchUnique(0);
  TestSketchUnique(0.5);
}

// if with_error is true, the test tolerates floating point error
void TestQuantileElemRank(int32_t device, Span<SketchEntry const> in,
                          Span<bst_row_t const> d_columns_ptr, bool with_error = false) {
  dh::LaunchN(device, in.size(), [=]XGBOOST_DEVICE(size_t idx) {
    auto column_id = dh::SegmentId(d_columns_ptr, idx);
    auto in_column = in.subspan(d_columns_ptr[column_id],
                                d_columns_ptr[column_id + 1] -
                                    d_columns_ptr[column_id]);
    auto constexpr kEps = 1e-6f;
    idx -= d_columns_ptr[column_id];
    float prev_rmin = idx == 0 ? 0.0f : in_column[idx-1].rmin;
    float prev_rmax = idx == 0 ? 0.0f : in_column[idx-1].rmax;
    float rmin_next = in_column[idx].RMinNext();

    if (with_error) {
      SPAN_CHECK(in_column[idx].rmin + in_column[idx].rmin * kEps >= prev_rmin);
      SPAN_CHECK(in_column[idx].rmax + in_column[idx].rmin * kEps >= prev_rmax);
      SPAN_CHECK(in_column[idx].rmax + in_column[idx].rmin * kEps >= rmin_next);
    } else {
      SPAN_CHECK(in_column[idx].rmin >= prev_rmin);
      SPAN_CHECK(in_column[idx].rmax >= prev_rmax);
      SPAN_CHECK(in_column[idx].rmax >= rmin_next);
    }
  });
  // Force sync to terminate current test instead of a later one.
  dh::DebugSyncDevice(__FILE__, __LINE__);
}


TEST(GPUQuantile, Prune) {
  constexpr size_t kRows = 1000, kCols = 100;
  RunWithSeedsAndBins(kRows, [=](int32_t seed, size_t n_bins, MetaInfo const& info) {
    SketchContainer sketch(n_bins, kCols, kRows, 0);

    HostDeviceVector<float> storage;
    std::string interface_str = RandomDataGenerator{kRows, kCols, 0}
                                    .Device(0)
                                    .Seed(seed)
                                    .GenerateArrayInterface(&storage);
    data::CupyAdapter adapter(interface_str);
    AdapterDeviceSketch(adapter.Value(), n_bins, info,
                        std::numeric_limits<float>::quiet_NaN(), &sketch);
    auto n_cuts = detail::RequiredSampleCutsPerColumn(n_bins, kRows);
    ASSERT_EQ(sketch.Data().size(), n_cuts * kCols);

    sketch.Prune(n_bins);
    if (n_bins <= kRows) {
      ASSERT_EQ(sketch.Data().size(), n_bins * kCols);
    } else {
      // LE because kRows * kCols is pushed into sketch, after removing duplicated entries
      // we might not have that much inputs for prune.
      ASSERT_LE(sketch.Data().size(), kRows * kCols);
    }
    // This is not necessarily true for all inputs without calling unique after
    // prune.
    ASSERT_TRUE(thrust::is_sorted(thrust::device, sketch.Data().data(),
                                  sketch.Data().data() + sketch.Data().size(),
                                  detail::SketchUnique{}));
    TestQuantileElemRank(0, sketch.Data(), sketch.ColumnsPtr());
  });
}

TEST(GPUQuantile, MergeEmpty) {
  constexpr size_t kRows = 1000, kCols = 100;
  size_t n_bins = 10;
  SketchContainer sketch_0(n_bins, kCols, kRows, 0);
  HostDeviceVector<float> storage_0;
  std::string interface_str_0 =
      RandomDataGenerator{kRows, kCols, 0}.Device(0).GenerateArrayInterface(
          &storage_0);
  data::CupyAdapter adapter_0(interface_str_0);
  MetaInfo info;
  AdapterDeviceSketch(adapter_0.Value(), n_bins, info,
                      std::numeric_limits<float>::quiet_NaN(), &sketch_0);

  std::vector<SketchEntry> entries_before(sketch_0.Data().size());
  dh::CopyDeviceSpanToVector(&entries_before, sketch_0.Data());
  std::vector<bst_row_t> ptrs_before(sketch_0.ColumnsPtr().size());
  dh::CopyDeviceSpanToVector(&ptrs_before, sketch_0.ColumnsPtr());
  thrust::device_vector<size_t> columns_ptr(kCols + 1);
  // Merge an empty sketch
  sketch_0.Merge(dh::ToSpan(columns_ptr), Span<SketchEntry>{});

  std::vector<SketchEntry> entries_after(sketch_0.Data().size());
  dh::CopyDeviceSpanToVector(&entries_after, sketch_0.Data());
  std::vector<bst_row_t> ptrs_after(sketch_0.ColumnsPtr().size());
  dh::CopyDeviceSpanToVector(&ptrs_after, sketch_0.ColumnsPtr());

  CHECK_EQ(entries_before.size(), entries_after.size());
  CHECK_EQ(ptrs_before.size(), ptrs_after.size());
  for (size_t i = 0; i < entries_before.size(); ++i) {
    CHECK_EQ(entries_before[i].value, entries_after[i].value);
    CHECK_EQ(entries_before[i].rmin, entries_after[i].rmin);
    CHECK_EQ(entries_before[i].rmax, entries_after[i].rmax);
    CHECK_EQ(entries_before[i].wmin, entries_after[i].wmin);
  }
  for (size_t i = 0; i < ptrs_before.size(); ++i) {
    CHECK_EQ(ptrs_before[i], ptrs_after[i]);
  }
}

TEST(GPUQuantile, MergeBasic) {
  constexpr size_t kRows = 1000, kCols = 100;
  RunWithSeedsAndBins(kRows, [=](int32_t seed, size_t n_bins, MetaInfo const& info) {
    SketchContainer sketch_0(n_bins, kCols, kRows, 0);
    HostDeviceVector<float> storage_0;
    std::string interface_str_0 = RandomDataGenerator{kRows, kCols, 0}
                                      .Device(0)
                                      .Seed(seed)
                                      .GenerateArrayInterface(&storage_0);
    data::CupyAdapter adapter_0(interface_str_0);
    AdapterDeviceSketch(adapter_0.Value(), n_bins, info,
                        std::numeric_limits<float>::quiet_NaN(), &sketch_0);

    SketchContainer sketch_1(n_bins, kCols, kRows * kRows, 0);
    HostDeviceVector<float> storage_1;
    std::string interface_str_1 = RandomDataGenerator{kRows, kCols, 0}
                                      .Device(0)
                                      .Seed(seed)
                                      .GenerateArrayInterface(&storage_1);
    data::CupyAdapter adapter_1(interface_str_1);
    AdapterDeviceSketch(adapter_1.Value(), n_bins, info,
                        std::numeric_limits<float>::quiet_NaN(), &sketch_1);

    size_t size_before_merge = sketch_0.Data().size();
    sketch_0.Merge(sketch_1.ColumnsPtr(), sketch_1.Data());
    if (info.weights_.Size() != 0) {
      TestQuantileElemRank(0, sketch_0.Data(), sketch_0.ColumnsPtr(), true);
      sketch_0.FixError();
      TestQuantileElemRank(0, sketch_0.Data(), sketch_0.ColumnsPtr(), false);
    } else {
      TestQuantileElemRank(0, sketch_0.Data(), sketch_0.ColumnsPtr());
    }

    auto columns_ptr = sketch_0.ColumnsPtr();
    std::vector<bst_row_t> h_columns_ptr(columns_ptr.size());
    dh::CopyDeviceSpanToVector(&h_columns_ptr, columns_ptr);
    ASSERT_EQ(h_columns_ptr.back(), sketch_1.Data().size() + size_before_merge);

    sketch_0.Unique();
    ASSERT_TRUE(
        thrust::is_sorted(thrust::device, sketch_0.Data().data(),
                          sketch_0.Data().data() + sketch_0.Data().size(),
                          detail::SketchUnique{}));
  });
}

void TestMergeDuplicated(int32_t n_bins, size_t cols, size_t rows, float frac) {
  MetaInfo info;
  int32_t seed = 0;
  SketchContainer sketch_0(n_bins, cols, rows, 0);
  HostDeviceVector<float> storage_0;
  std::string interface_str_0 = RandomDataGenerator{rows, cols, 0}
                                    .Device(0)
                                    .Seed(seed)
                                    .GenerateArrayInterface(&storage_0);
  data::CupyAdapter adapter_0(interface_str_0);
  AdapterDeviceSketch(adapter_0.Value(), n_bins, info,
                      std::numeric_limits<float>::quiet_NaN(),
                      &sketch_0);

  size_t f_rows = rows * frac;
  SketchContainer sketch_1(n_bins, cols, f_rows, 0);
  HostDeviceVector<float> storage_1;
  std::string interface_str_1 = RandomDataGenerator{f_rows, cols, 0}
                                    .Device(0)
                                    .Seed(seed)
                                    .GenerateArrayInterface(&storage_1);
  auto data_1 = storage_1.DeviceSpan();
  auto tuple_it = thrust::make_tuple(
      thrust::make_counting_iterator<size_t>(0ul), data_1.data());
  using Tuple = thrust::tuple<size_t, float>;
  auto it = thrust::make_zip_iterator(tuple_it);
  thrust::transform(thrust::device, it, it + data_1.size(), data_1.data(),
                    [=] __device__(Tuple const &tuple) {
                      auto i = thrust::get<0>(tuple);
                      if (thrust::get<0>(tuple) % 2 == 0) {
                        return 0.0f;
                      } else {
                        return thrust::get<1>(tuple);
                      }
                    });
  data::CupyAdapter adapter_1(interface_str_1);
  AdapterDeviceSketch(adapter_1.Value(), n_bins, info,
                      std::numeric_limits<float>::quiet_NaN(),
                      &sketch_1);

  size_t size_before_merge = sketch_0.Data().size();
  sketch_0.Merge(sketch_1.ColumnsPtr(), sketch_1.Data());
  TestQuantileElemRank(0, sketch_0.Data(), sketch_0.ColumnsPtr());

  auto columns_ptr = sketch_0.ColumnsPtr();
  std::vector<bst_row_t> h_columns_ptr(columns_ptr.size());
  dh::CopyDeviceSpanToVector(&h_columns_ptr, columns_ptr);
  ASSERT_EQ(h_columns_ptr.back(), sketch_1.Data().size() + size_before_merge);

  sketch_0.Unique();
  ASSERT_TRUE(thrust::is_sorted(thrust::device, sketch_0.Data().data(),
                                sketch_0.Data().data() + sketch_0.Data().size(),
                                detail::SketchUnique{}));
}

TEST(GPUQuantile, MergeDuplicated) {
  size_t n_bins = 256;
  constexpr size_t kRows = 1000, kCols = 100;
  for (float frac = 0.5; frac < 2.5; frac += 0.5) {
    TestMergeDuplicated(n_bins, kRows, kCols, frac);
  }
}

TEST(GPUQuantile, AllReduceBasic) {
  // This test is supposed to run by a python test that setups the environment.
  std::string msg {"Skipping AllReduce test"};
#if defined(__linux__) && defined(XGBOOST_USE_NCCL)
  auto n_gpus = AllVisibleGPUs();
  InitRabitContext(msg, n_gpus);
  auto world = rabit::GetWorldSize();
  if (world != 1) {
    ASSERT_EQ(world, n_gpus);
  } else {
    return;
  }

  constexpr size_t kRows = 1000, kCols = 100;
  RunWithSeedsAndBins(kRows, [=](int32_t seed, size_t n_bins, MetaInfo const& info) {
    // Set up single node version;
    SketchContainer sketch_on_single_node(n_bins, kCols, kRows, 0);

    size_t intermediate_num_cuts =
        std::min(kRows * world, static_cast<size_t>(n_bins * WQSketch::kFactor));
    std::vector<SketchContainer> containers;
    for (auto rank = 0; rank < world; ++rank) {
      HostDeviceVector<float> storage;
      std::string interface_str = RandomDataGenerator{kRows, kCols, 0}
                                      .Device(0)
                                      .Seed(rank + seed)
                                      .GenerateArrayInterface(&storage);
      data::CupyAdapter adapter(interface_str);
      containers.emplace_back(n_bins, kCols, kRows, 0);
      AdapterDeviceSketch(adapter.Value(), n_bins, info,
                          std::numeric_limits<float>::quiet_NaN(),
                          &containers.back());
    }
    for (auto& sketch : containers) {
      sketch.Prune(intermediate_num_cuts);
      sketch_on_single_node.Merge(sketch.ColumnsPtr(), sketch.Data());
      sketch_on_single_node.FixError();
    }
    sketch_on_single_node.Unique();
    TestQuantileElemRank(0, sketch_on_single_node.Data(),
                         sketch_on_single_node.ColumnsPtr());

    // Set up distributed version.  We rely on using rank as seed to generate
    // the exact same copy of data.
    auto rank = rabit::GetRank();
    SketchContainer sketch_distributed(n_bins, kCols, kRows, 0);
    HostDeviceVector<float> storage;
    std::string interface_str = RandomDataGenerator{kRows, kCols, 0}
                                    .Device(0)
                                    .Seed(rank + seed)
                                    .GenerateArrayInterface(&storage);
    data::CupyAdapter adapter(interface_str);
    AdapterDeviceSketch(adapter.Value(), n_bins, info,
                        std::numeric_limits<float>::quiet_NaN(),
                        &sketch_distributed);
    sketch_distributed.AllReduce();
    sketch_distributed.Unique();

    ASSERT_EQ(sketch_distributed.ColumnsPtr().size(),
              sketch_on_single_node.ColumnsPtr().size());
    ASSERT_EQ(sketch_distributed.Data().size(),
              sketch_on_single_node.Data().size());

    TestQuantileElemRank(0, sketch_distributed.Data(),
                         sketch_distributed.ColumnsPtr());

    std::vector<SketchEntry> single_node_data(
        sketch_on_single_node.Data().size());
    dh::CopyDeviceSpanToVector(&single_node_data, sketch_on_single_node.Data());

    std::vector<SketchEntry> distributed_data(sketch_distributed.Data().size());
    dh::CopyDeviceSpanToVector(&distributed_data, sketch_distributed.Data());
    float Eps = 2e-4 * world;

    for (size_t i = 0; i < single_node_data.size(); ++i) {
      ASSERT_NEAR(single_node_data[i].value, distributed_data[i].value, Eps);
      ASSERT_NEAR(single_node_data[i].rmax, distributed_data[i].rmax, Eps);
      ASSERT_NEAR(single_node_data[i].rmin, distributed_data[i].rmin, Eps);
      ASSERT_NEAR(single_node_data[i].wmin, distributed_data[i].wmin, Eps);
    }
  });
  rabit::Finalize();
#else
  LOG(WARNING) << msg;
  return;
#endif  // !defined(__linux__) && defined(XGBOOST_USE_NCCL)
}

TEST(GPUQuantile, SameOnAllWorkers) {
  std::string msg {"Skipping SameOnAllWorkers test"};
#if defined(__linux__) && defined(XGBOOST_USE_NCCL)
  auto n_gpus = AllVisibleGPUs();
  InitRabitContext(msg, n_gpus);
  auto world = rabit::GetWorldSize();
  if (world != 1) {
    ASSERT_EQ(world, n_gpus);
  } else {
    return;
  }

  constexpr size_t kRows = 1000, kCols = 100;
  RunWithSeedsAndBins(kRows, [=](int32_t seed, size_t n_bins,
                                 MetaInfo const &info) {
    auto rank = rabit::GetRank();
    SketchContainer sketch_distributed(n_bins, kCols, kRows, 0);
    HostDeviceVector<float> storage;
    std::string interface_str = RandomDataGenerator{kRows, kCols, 0}
                                    .Device(0)
                                    .Seed(rank + seed)
                                    .GenerateArrayInterface(&storage);
    data::CupyAdapter adapter(interface_str);
    AdapterDeviceSketch(adapter.Value(), n_bins, info,
                        std::numeric_limits<float>::quiet_NaN(),
                        &sketch_distributed);
    sketch_distributed.AllReduce();
    sketch_distributed.Unique();
    TestQuantileElemRank(0, sketch_distributed.Data(), sketch_distributed.ColumnsPtr());

    // Test for all workers having the same sketch.
    size_t n_data = sketch_distributed.Data().size();
    rabit::Allreduce<rabit::op::Max>(&n_data, 1);
    ASSERT_EQ(n_data, sketch_distributed.Data().size());
    size_t size_as_float =
        sketch_distributed.Data().size_bytes() / sizeof(float);
    auto local_data = Span<float const>{
        reinterpret_cast<float const *>(sketch_distributed.Data().data()),
        size_as_float};

    dh::caching_device_vector<float> all_workers(size_as_float * world);
    thrust::fill(all_workers.begin(), all_workers.end(), 0);
    thrust::copy(thrust::device, local_data.data(),
                 local_data.data() + local_data.size(),
                 all_workers.begin() + local_data.size() * rank);
    dh::AllReducer reducer;
    reducer.Init(0);

    reducer.AllReduceSum(all_workers.data().get(), all_workers.data().get(),
                         all_workers.size());
    reducer.Synchronize();

    auto base_line = dh::ToSpan(all_workers).subspan(0, size_as_float);
    std::vector<float> h_base_line(base_line.size());
    dh::CopyDeviceSpanToVector(&h_base_line, base_line);

    size_t offset = 0;
    for (size_t i = 0; i < world; ++i) {
      auto comp = dh::ToSpan(all_workers).subspan(offset, size_as_float);
      std::vector<float> h_comp(comp.size());
      dh::CopyDeviceSpanToVector(&h_comp, comp);
      ASSERT_EQ(comp.size(), base_line.size());
      for (size_t j = 0; j < h_comp.size(); ++j) {
        ASSERT_NEAR(h_base_line[j], h_comp[j], kRtEps);
      }
      offset += size_as_float;
    }
  });
#else
  LOG(WARNING) << msg;
  return;
#endif  // !defined(__linux__) && defined(XGBOOST_USE_NCCL)
}
}  // namespace common
}  // namespace xgboost
