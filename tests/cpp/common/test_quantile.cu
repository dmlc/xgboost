#include <gtest/gtest.h>
#include "../../../src/common/quantile.cuh"

namespace xgboost {
namespace common {
TEST(GPUQuantile, Basic) {
  constexpr size_t kRows = 1000, kCols = 100, kBins = 256;
  SketchContainer sketch(kBins, kCols, kRows, 0);
  dh::device_vector<SketchEntry> entries;
}

TEST(GPUQuantile, Prune) {
  constexpr size_t kRows = 1000, kCols = 100, kBins = 256;
  SketchContainer sketch(kBins, kCols, kRows, 0);
}

TEST(GPUQuantile, Merge) {
  constexpr size_t kRows = 1000, kCols = 100, kBins = 256;
  SketchContainer sketch(kBins, kCols, kRows, 0);
}
}  // namespace common
}  // namespace xgboost
