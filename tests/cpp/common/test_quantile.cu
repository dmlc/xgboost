#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include "test_quantile.h"
#include "../helpers.h"
#include "../../../src/common/hist_util.cuh"
#include "../../../src/common/quantile.cuh"

namespace xgboost {
namespace {
struct IsSorted {
  XGBOOST_DEVICE bool operator()(common::SketchEntry const& a, common::SketchEntry const& b) const {
    return a.value < b.value;
  }
};
}
namespace common {
TEST(GPUQuantile, Basic) {
  constexpr size_t kRows = 1000, kCols = 100, kBins = 256;
  HostDeviceVector<FeatureType> ft;
  SketchContainer sketch(ft, kBins, kCols, kRows, 0);
  dh::caching_device_vector<Entry> entries;
  dh::device_vector<bst_row_t> cuts_ptr(kCols+1);
  thrust::fill(cuts_ptr.begin(), cuts_ptr.end(), 0);
  // Push empty
  sketch.Push(dh::ToSpan(entries), dh::ToSpan(cuts_ptr), dh::ToSpan(cuts_ptr), 0);
  ASSERT_EQ(sketch.Data().size(), 0);
}

void TestSketchUnique(float sparsity) {
  constexpr size_t kRows = 1000, kCols = 100;
  RunWithSeedsAndBins(kRows, [kRows, kCols, sparsity](int32_t seed, size_t n_bins, MetaInfo const& info) {
    HostDeviceVector<FeatureType> ft;
    SketchContainer sketch(ft, n_bins, kCols, kRows, 0);

    HostDeviceVector<float> storage;
    std::string interface_str = RandomDataGenerator{kRows, kCols, sparsity}
                                    .Seed(seed)
                                    .Device(0)
                                    .GenerateArrayInterface(&storage);
    data::CupyAdapter adapter(interface_str);
    AdapterDeviceSketch(adapter.Value(), n_bins, info,
                        std::numeric_limits<float>::quiet_NaN(), &sketch);
    auto n_cuts = detail::RequiredSampleCutsPerColumn(n_bins, kRows);

    dh::caching_device_vector<size_t> column_sizes_scan;
    HostDeviceVector<size_t> cut_sizes_scan;
    auto batch = adapter.Value();
    data::IsValidFunctor is_valid(std::numeric_limits<float>::quiet_NaN());
    auto batch_iter = dh::MakeTransformIterator<data::COOTuple>(
        thrust::make_counting_iterator(0llu),
        [=] __device__(size_t idx) { return batch.GetElement(idx); });
    auto end = kCols * kRows;
    detail::GetColumnSizesScan(0, kCols, n_cuts, batch_iter, is_valid, 0, end,
                               &cut_sizes_scan, &column_sizes_scan);
    auto const& cut_sizes = cut_sizes_scan.HostVector();
    ASSERT_LE(sketch.Data().size(), cut_sizes.back());

    std::vector<size_t> h_columns_ptr(sketch.ColumnsPtr().size());
    dh::CopyDeviceSpanToVector(&h_columns_ptr, sketch.ColumnsPtr());
    ASSERT_EQ(sketch.Data().size(), h_columns_ptr.back());

    sketch.Unique();

    std::vector<SketchEntry> h_data(sketch.Data().size());
    thrust::copy(dh::tcbegin(sketch.Data()), dh::tcend(sketch.Data()), h_data.begin());

    for (size_t i = 1; i < h_columns_ptr.size(); ++i) {
      auto begin = h_columns_ptr[i - 1];
      auto column = common::Span<SketchEntry>(h_data).subspan(begin, h_columns_ptr[i] - begin);
      ASSERT_TRUE(std::is_sorted(column.begin(), column.end(), IsSorted{}));
    }
  });
}

TEST(GPUQuantile, Unique) {
  TestSketchUnique(0);
  TestSketchUnique(0.5);
}

// if with_error is true, the test tolerates floating point error
void TestQuantileElemRank(int32_t device, Span<SketchEntry const> in,
                          Span<bst_row_t const> d_columns_ptr, bool with_error = false) {
  dh::safe_cuda(hipSetDevice(device));
  std::vector<SketchEntry> h_in(in.size());
  dh::CopyDeviceSpanToVector(&h_in, in);
  std::vector<bst_row_t> h_columns_ptr(d_columns_ptr.size());
  dh::CopyDeviceSpanToVector(&h_columns_ptr, d_columns_ptr);

  for (size_t i = 1; i < d_columns_ptr.size(); ++i) {
    auto column_id = i - 1;
    auto beg = h_columns_ptr[column_id];
    auto end = h_columns_ptr[i];

    auto in_column = Span<SketchEntry>{h_in}.subspan(beg, end - beg);
    for (size_t idx = 1; idx < in_column.size(); ++idx) {
      float prev_rmin = in_column[idx - 1].rmin;
      float prev_rmax = in_column[idx - 1].rmax;
      float rmin_next = in_column[idx].RMinNext();
      if (with_error) {
        ASSERT_GE(in_column[idx].rmin + in_column[idx].rmin * kRtEps,
                  prev_rmin);
        ASSERT_GE(in_column[idx].rmax + in_column[idx].rmin * kRtEps, prev_rmax);
        ASSERT_GE(in_column[idx].rmax + in_column[idx].rmin * kRtEps,
                  rmin_next);
      } else {
        ASSERT_GE(in_column[idx].rmin, prev_rmin);
        ASSERT_GE(in_column[idx].rmax, prev_rmax);
        ASSERT_GE(in_column[idx].rmax, rmin_next);
      }
    }
  }
}

TEST(GPUQuantile, Prune) {
  constexpr size_t kRows = 1000, kCols = 100;
  RunWithSeedsAndBins(kRows, [=](int32_t seed, size_t n_bins, MetaInfo const& info) {
    HostDeviceVector<FeatureType> ft;
    SketchContainer sketch(ft, n_bins, kCols, kRows, 0);

    HostDeviceVector<float> storage;
    std::string interface_str = RandomDataGenerator{kRows, kCols, 0}
                                    .Device(0)
                                    .Seed(seed)
                                    .GenerateArrayInterface(&storage);
    data::CupyAdapter adapter(interface_str);
    AdapterDeviceSketch(adapter.Value(), n_bins, info,
                        std::numeric_limits<float>::quiet_NaN(), &sketch);
    auto n_cuts = detail::RequiredSampleCutsPerColumn(n_bins, kRows);
    // LE because kRows * kCols is pushed into sketch, after removing
    // duplicated entries we might not have that much inputs for prune.
    ASSERT_LE(sketch.Data().size(), n_cuts * kCols);

    sketch.Prune(n_bins);
    ASSERT_LE(sketch.Data().size(), kRows * kCols);
    // This is not necessarily true for all inputs without calling unique after
    // prune.
    ASSERT_TRUE(thrust::is_sorted(thrust::device, sketch.Data().data(),
                                  sketch.Data().data() + sketch.Data().size(),
                                  detail::SketchUnique{}));
    TestQuantileElemRank(0, sketch.Data(), sketch.ColumnsPtr());
  });
}

TEST(GPUQuantile, MergeEmpty) {
  constexpr size_t kRows = 1000, kCols = 100;
  size_t n_bins = 10;
  HostDeviceVector<FeatureType> ft;
  SketchContainer sketch_0(ft, n_bins, kCols, kRows, 0);
  HostDeviceVector<float> storage_0;
  std::string interface_str_0 =
      RandomDataGenerator{kRows, kCols, 0}.Device(0).GenerateArrayInterface(
          &storage_0);
  data::CupyAdapter adapter_0(interface_str_0);
  MetaInfo info;
  AdapterDeviceSketch(adapter_0.Value(), n_bins, info,
                      std::numeric_limits<float>::quiet_NaN(), &sketch_0);

  std::vector<SketchEntry> entries_before(sketch_0.Data().size());
  dh::CopyDeviceSpanToVector(&entries_before, sketch_0.Data());
  std::vector<bst_row_t> ptrs_before(sketch_0.ColumnsPtr().size());
  dh::CopyDeviceSpanToVector(&ptrs_before, sketch_0.ColumnsPtr());
  thrust::device_vector<size_t> columns_ptr(kCols + 1);
  // Merge an empty sketch
  sketch_0.Merge(dh::ToSpan(columns_ptr), Span<SketchEntry>{});

  std::vector<SketchEntry> entries_after(sketch_0.Data().size());
  dh::CopyDeviceSpanToVector(&entries_after, sketch_0.Data());
  std::vector<bst_row_t> ptrs_after(sketch_0.ColumnsPtr().size());
  dh::CopyDeviceSpanToVector(&ptrs_after, sketch_0.ColumnsPtr());

  CHECK_EQ(entries_before.size(), entries_after.size());
  CHECK_EQ(ptrs_before.size(), ptrs_after.size());
  for (size_t i = 0; i < entries_before.size(); ++i) {
    CHECK_EQ(entries_before[i].value, entries_after[i].value);
    CHECK_EQ(entries_before[i].rmin, entries_after[i].rmin);
    CHECK_EQ(entries_before[i].rmax, entries_after[i].rmax);
    CHECK_EQ(entries_before[i].wmin, entries_after[i].wmin);
  }
  for (size_t i = 0; i < ptrs_before.size(); ++i) {
    CHECK_EQ(ptrs_before[i], ptrs_after[i]);
  }
}

TEST(GPUQuantile, MergeBasic) {
  constexpr size_t kRows = 1000, kCols = 100;
  RunWithSeedsAndBins(kRows, [=](int32_t seed, size_t n_bins, MetaInfo const &info) {
    HostDeviceVector<FeatureType> ft;
    SketchContainer sketch_0(ft, n_bins, kCols, kRows, 0);
    HostDeviceVector<float> storage_0;
    std::string interface_str_0 = RandomDataGenerator{kRows, kCols, 0}
                                      .Device(0)
                                      .Seed(seed)
                                      .GenerateArrayInterface(&storage_0);
    data::CupyAdapter adapter_0(interface_str_0);
    AdapterDeviceSketch(adapter_0.Value(), n_bins, info,
                        std::numeric_limits<float>::quiet_NaN(), &sketch_0);

    SketchContainer sketch_1(ft, n_bins, kCols, kRows * kRows, 0);
    HostDeviceVector<float> storage_1;
    std::string interface_str_1 = RandomDataGenerator{kRows, kCols, 0}
                                      .Device(0)
                                      .Seed(seed)
                                      .GenerateArrayInterface(&storage_1);
    data::CupyAdapter adapter_1(interface_str_1);
    AdapterDeviceSketch(adapter_1.Value(), n_bins, info,
                        std::numeric_limits<float>::quiet_NaN(), &sketch_1);

    size_t size_before_merge = sketch_0.Data().size();
    sketch_0.Merge(sketch_1.ColumnsPtr(), sketch_1.Data());
    if (info.weights_.Size() != 0) {
      TestQuantileElemRank(0, sketch_0.Data(), sketch_0.ColumnsPtr(), true);
      sketch_0.FixError();
      TestQuantileElemRank(0, sketch_0.Data(), sketch_0.ColumnsPtr(), false);
    } else {
      TestQuantileElemRank(0, sketch_0.Data(), sketch_0.ColumnsPtr());
    }

    auto columns_ptr = sketch_0.ColumnsPtr();
    std::vector<bst_row_t> h_columns_ptr(columns_ptr.size());
    dh::CopyDeviceSpanToVector(&h_columns_ptr, columns_ptr);
    ASSERT_EQ(h_columns_ptr.back(), sketch_1.Data().size() + size_before_merge);

    sketch_0.Unique();
    ASSERT_TRUE(
        thrust::is_sorted(thrust::device, sketch_0.Data().data(),
                          sketch_0.Data().data() + sketch_0.Data().size(),
                          detail::SketchUnique{}));
  });
}

void TestMergeDuplicated(int32_t n_bins, size_t cols, size_t rows, float frac) {
  MetaInfo info;
  int32_t seed = 0;
  HostDeviceVector<FeatureType> ft;
  SketchContainer sketch_0(ft, n_bins, cols, rows, 0);
  HostDeviceVector<float> storage_0;
  std::string interface_str_0 = RandomDataGenerator{rows, cols, 0}
                                    .Device(0)
                                    .Seed(seed)
                                    .GenerateArrayInterface(&storage_0);
  data::CupyAdapter adapter_0(interface_str_0);
  AdapterDeviceSketch(adapter_0.Value(), n_bins, info,
                      std::numeric_limits<float>::quiet_NaN(),
                      &sketch_0);

  size_t f_rows = rows * frac;
  SketchContainer sketch_1(ft, n_bins, cols, f_rows, 0);
  HostDeviceVector<float> storage_1;
  std::string interface_str_1 = RandomDataGenerator{f_rows, cols, 0}
                                    .Device(0)
                                    .Seed(seed)
                                    .GenerateArrayInterface(&storage_1);
  auto data_1 = storage_1.DeviceSpan();
  auto tuple_it = thrust::make_tuple(
      thrust::make_counting_iterator<size_t>(0ul), data_1.data());
  using Tuple = thrust::tuple<size_t, float>;
  auto it = thrust::make_zip_iterator(tuple_it);
  thrust::transform(thrust::device, it, it + data_1.size(), data_1.data(),
                    [=] __device__(Tuple const &tuple) {
                      auto i = thrust::get<0>(tuple);
                      if (thrust::get<0>(tuple) % 2 == 0) {
                        return 0.0f;
                      } else {
                        return thrust::get<1>(tuple);
                      }
                    });
  data::CupyAdapter adapter_1(interface_str_1);
  AdapterDeviceSketch(adapter_1.Value(), n_bins, info,
                      std::numeric_limits<float>::quiet_NaN(),
                      &sketch_1);

  size_t size_before_merge = sketch_0.Data().size();
  sketch_0.Merge(sketch_1.ColumnsPtr(), sketch_1.Data());
  TestQuantileElemRank(0, sketch_0.Data(), sketch_0.ColumnsPtr());

  auto columns_ptr = sketch_0.ColumnsPtr();
  std::vector<bst_row_t> h_columns_ptr(columns_ptr.size());
  dh::CopyDeviceSpanToVector(&h_columns_ptr, columns_ptr);
  ASSERT_EQ(h_columns_ptr.back(), sketch_1.Data().size() + size_before_merge);

  sketch_0.Unique();
  columns_ptr = sketch_0.ColumnsPtr();
  dh::CopyDeviceSpanToVector(&h_columns_ptr, columns_ptr);

  std::vector<SketchEntry> h_data(sketch_0.Data().size());
  dh::CopyDeviceSpanToVector(&h_data, sketch_0.Data());
  for (size_t i = 1; i < h_columns_ptr.size(); ++i) {
    auto begin = h_columns_ptr[i - 1];
    auto column = Span<SketchEntry> {h_data}.subspan(begin, h_columns_ptr[i] - begin);
    ASSERT_TRUE(std::is_sorted(column.begin(), column.end(), IsSorted{}));
  }
}

TEST(GPUQuantile, MergeDuplicated) {
  size_t n_bins = 256;
  constexpr size_t kRows = 1000, kCols = 100;
  for (float frac = 0.5; frac < 2.5; frac += 0.5) {
    TestMergeDuplicated(n_bins, kRows, kCols, frac);
  }
}

TEST(GPUQuantile, MultiMerge) {
  constexpr size_t kRows = 20, kCols = 1;
  int32_t world = 2;
  RunWithSeedsAndBins(kRows, [=](int32_t seed, size_t n_bins,
                                 MetaInfo const &info) {
    // Set up single node version
    HostDeviceVector<FeatureType> ft;
    SketchContainer sketch_on_single_node(ft, n_bins, kCols, kRows, 0);

    size_t intermediate_num_cuts = std::min(
        kRows * world, static_cast<size_t>(n_bins * WQSketch::kFactor));
    std::vector<SketchContainer> containers;
    for (auto rank = 0; rank < world; ++rank) {
      HostDeviceVector<float> storage;
      std::string interface_str = RandomDataGenerator{kRows, kCols, 0}
                                      .Device(0)
                                      .Seed(rank + seed)
                                      .GenerateArrayInterface(&storage);
      data::CupyAdapter adapter(interface_str);
      HostDeviceVector<FeatureType> ft;
      containers.emplace_back(ft, n_bins, kCols, kRows, 0);
      AdapterDeviceSketch(adapter.Value(), n_bins, info,
                          std::numeric_limits<float>::quiet_NaN(),
                          &containers.back());
    }
    for (auto &sketch : containers) {
      sketch.Prune(intermediate_num_cuts);
      sketch_on_single_node.Merge(sketch.ColumnsPtr(), sketch.Data());
      sketch_on_single_node.FixError();
    }
    TestQuantileElemRank(0, sketch_on_single_node.Data(),
                         sketch_on_single_node.ColumnsPtr());

    sketch_on_single_node.Unique();
    TestQuantileElemRank(0, sketch_on_single_node.Data(),
                         sketch_on_single_node.ColumnsPtr());
  });
}

TEST(GPUQuantile, AllReduceBasic) {
  // This test is supposed to run by a python test that setups the environment.
  std::string msg {"Skipping AllReduce test"};
  auto n_gpus = AllVisibleGPUs();
  InitRabitContext(msg, n_gpus);
  auto world = rabit::GetWorldSize();
  if (world != 1) {
    ASSERT_EQ(world, n_gpus);
  } else {
    return;
  }

  constexpr size_t kRows = 1000, kCols = 100;
  RunWithSeedsAndBins(kRows, [=](int32_t seed, size_t n_bins, MetaInfo const& info) {
    // Set up single node version;
    HostDeviceVector<FeatureType> ft;
    SketchContainer sketch_on_single_node(ft, n_bins, kCols, kRows, 0);

    size_t intermediate_num_cuts = std::min(
        kRows * world, static_cast<size_t>(n_bins * WQSketch::kFactor));
    std::vector<SketchContainer> containers;
    for (auto rank = 0; rank < world; ++rank) {
      HostDeviceVector<float> storage;
      std::string interface_str = RandomDataGenerator{kRows, kCols, 0}
                                      .Device(0)
                                      .Seed(rank + seed)
                                      .GenerateArrayInterface(&storage);
      data::CupyAdapter adapter(interface_str);
      HostDeviceVector<FeatureType> ft;
      containers.emplace_back(ft, n_bins, kCols, kRows, 0);
      AdapterDeviceSketch(adapter.Value(), n_bins, info,
                          std::numeric_limits<float>::quiet_NaN(),
                          &containers.back());
    }
    for (auto &sketch : containers) {
      sketch.Prune(intermediate_num_cuts);
      sketch_on_single_node.Merge(sketch.ColumnsPtr(), sketch.Data());
      sketch_on_single_node.FixError();
    }
    sketch_on_single_node.Unique();
    TestQuantileElemRank(0, sketch_on_single_node.Data(),
                         sketch_on_single_node.ColumnsPtr());

    // Set up distributed version.  We rely on using rank as seed to generate
    // the exact same copy of data.
    auto rank = rabit::GetRank();
    SketchContainer sketch_distributed(ft, n_bins, kCols, kRows, 0);
    HostDeviceVector<float> storage;
    std::string interface_str = RandomDataGenerator{kRows, kCols, 0}
                                    .Device(0)
                                    .Seed(rank + seed)
                                    .GenerateArrayInterface(&storage);
    data::CupyAdapter adapter(interface_str);
    AdapterDeviceSketch(adapter.Value(), n_bins, info,
                        std::numeric_limits<float>::quiet_NaN(),
                        &sketch_distributed);
    sketch_distributed.AllReduce();
    sketch_distributed.Unique();

    ASSERT_EQ(sketch_distributed.ColumnsPtr().size(),
              sketch_on_single_node.ColumnsPtr().size());
    ASSERT_EQ(sketch_distributed.Data().size(),
              sketch_on_single_node.Data().size());

    TestQuantileElemRank(0, sketch_distributed.Data(),
                         sketch_distributed.ColumnsPtr());

    std::vector<SketchEntry> single_node_data(
        sketch_on_single_node.Data().size());
    dh::CopyDeviceSpanToVector(&single_node_data, sketch_on_single_node.Data());

    std::vector<SketchEntry> distributed_data(sketch_distributed.Data().size());
    dh::CopyDeviceSpanToVector(&distributed_data, sketch_distributed.Data());
    float Eps = 2e-4 * world;

    for (size_t i = 0; i < single_node_data.size(); ++i) {
      ASSERT_NEAR(single_node_data[i].value, distributed_data[i].value, Eps);
      ASSERT_NEAR(single_node_data[i].rmax, distributed_data[i].rmax, Eps);
      ASSERT_NEAR(single_node_data[i].rmin, distributed_data[i].rmin, Eps);
      ASSERT_NEAR(single_node_data[i].wmin, distributed_data[i].wmin, Eps);
    }
  });
  rabit::Finalize();
}

TEST(GPUQuantile, SameOnAllWorkers) {
  std::string msg {"Skipping SameOnAllWorkers test"};
  auto n_gpus = AllVisibleGPUs();
  InitRabitContext(msg, n_gpus);
  auto world = rabit::GetWorldSize();
  if (world != 1) {
    ASSERT_EQ(world, n_gpus);
  } else {
    return;
  }

  constexpr size_t kRows = 1000, kCols = 100;
  RunWithSeedsAndBins(kRows, [=](int32_t seed, size_t n_bins,
                                 MetaInfo const &info) {
    auto rank = rabit::GetRank();
    HostDeviceVector<FeatureType> ft;
    SketchContainer sketch_distributed(ft, n_bins, kCols, kRows, 0);
    HostDeviceVector<float> storage;
    std::string interface_str = RandomDataGenerator{kRows, kCols, 0}
                                    .Device(0)
                                    .Seed(rank + seed)
                                    .GenerateArrayInterface(&storage);
    data::CupyAdapter adapter(interface_str);
    AdapterDeviceSketch(adapter.Value(), n_bins, info,
                        std::numeric_limits<float>::quiet_NaN(),
                        &sketch_distributed);
    sketch_distributed.AllReduce();
    sketch_distributed.Unique();
    TestQuantileElemRank(0, sketch_distributed.Data(), sketch_distributed.ColumnsPtr());

    // Test for all workers having the same sketch.
    size_t n_data = sketch_distributed.Data().size();
    rabit::Allreduce<rabit::op::Max>(&n_data, 1);
    ASSERT_EQ(n_data, sketch_distributed.Data().size());
    size_t size_as_float =
        sketch_distributed.Data().size_bytes() / sizeof(float);
    auto local_data = Span<float const>{
        reinterpret_cast<float const *>(sketch_distributed.Data().data()),
        size_as_float};

    dh::caching_device_vector<float> all_workers(size_as_float * world);
    thrust::fill(all_workers.begin(), all_workers.end(), 0);
    thrust::copy(thrust::device, local_data.data(),
                 local_data.data() + local_data.size(),
                 all_workers.begin() + local_data.size() * rank);
    dh::AllReducer reducer;
    reducer.Init(0);

    reducer.AllReduceSum(all_workers.data().get(), all_workers.data().get(),
                         all_workers.size());
    reducer.Synchronize();

    auto base_line = dh::ToSpan(all_workers).subspan(0, size_as_float);
    std::vector<float> h_base_line(base_line.size());
    dh::CopyDeviceSpanToVector(&h_base_line, base_line);

    size_t offset = 0;
    for (decltype(world) i = 0; i < world; ++i) {
      auto comp = dh::ToSpan(all_workers).subspan(offset, size_as_float);
      std::vector<float> h_comp(comp.size());
      dh::CopyDeviceSpanToVector(&h_comp, comp);
      ASSERT_EQ(comp.size(), base_line.size());
      for (size_t j = 0; j < h_comp.size(); ++j) {
        ASSERT_NEAR(h_base_line[j], h_comp[j], kRtEps);
      }
      offset += size_as_float;
    }
  });
}

TEST(GPUQuantile, Push) {
  size_t constexpr kRows = 100;
  std::vector<float> data(kRows);

  std::fill(data.begin(), data.begin() + (data.size() / 2), 0.3f);
  std::fill(data.begin() + (data.size() / 2), data.end(), 0.5f);
  int32_t n_bins = 128;
  bst_feature_t constexpr kCols = 1;

  std::vector<Entry> entries(kRows);
  for (bst_feature_t i = 0; i < entries.size(); ++i) {
    Entry e{i, data[i]};
    entries[i] = e;
  }

  dh::device_vector<Entry> d_entries(entries);
  dh::device_vector<size_t> columns_ptr(2);
  columns_ptr[0] = 0;
  columns_ptr[1] = kRows;

  HostDeviceVector<FeatureType> ft;
  SketchContainer sketch(ft, n_bins, kCols, kRows, 0);
  sketch.Push(dh::ToSpan(d_entries), dh::ToSpan(columns_ptr), dh::ToSpan(columns_ptr), kRows, {});

  auto sketch_data = sketch.Data();

  thrust::host_vector<SketchEntry> h_sketch_data(sketch_data.size());

  auto ptr = thrust::device_ptr<SketchEntry const>(sketch_data.data());
  thrust::copy(ptr, ptr + sketch_data.size(), h_sketch_data.begin());
  ASSERT_EQ(h_sketch_data.size(), 2);

  auto v_0 = h_sketch_data[0];
  ASSERT_EQ(v_0.rmin, 0);
  ASSERT_EQ(v_0.wmin, kRows / 2.0f);
  ASSERT_EQ(v_0.rmax, kRows / 2.0f);

  auto v_1 = h_sketch_data[1];
  ASSERT_EQ(v_1.rmin, kRows / 2.0f);
  ASSERT_EQ(v_1.wmin, kRows / 2.0f);
  ASSERT_EQ(v_1.rmax, static_cast<float>(kRows));
}

TEST(GPUQuantile, MultiColPush) {
  size_t constexpr kRows = 100, kCols = 4;
  std::vector<float> data(kRows * kCols);

  std::fill(data.begin(), data.begin() + (data.size() / 2), 0.3f);

  std::vector<Entry> entries(kRows * kCols);

  for (bst_feature_t c = 0; c < kCols; ++c) {
    for (size_t r = 0; r < kRows; ++r) {
      float v = (r >= kRows / 2) ? 0.7 : 0.4;
      auto e = Entry{c, v};
      entries[c * kRows + r] = e;
    }
  }

  int32_t n_bins = 16;
  HostDeviceVector<FeatureType> ft;
  SketchContainer sketch(ft, n_bins, kCols, kRows, 0);
  dh::device_vector<Entry> d_entries {entries};

  dh::device_vector<size_t> columns_ptr(kCols + 1, 0);
  for (size_t i = 1; i < kCols + 1; ++i) {
    columns_ptr[i] = kRows;
  }
  thrust::inclusive_scan(thrust::device, columns_ptr.begin(), columns_ptr.end(),
                         columns_ptr.begin());
  dh::device_vector<size_t> cuts_ptr(columns_ptr);

  sketch.Push(dh::ToSpan(d_entries), dh::ToSpan(columns_ptr),
              dh::ToSpan(cuts_ptr), kRows * kCols, {});

  auto sketch_data = sketch.Data();
  ASSERT_EQ(sketch_data.size(), kCols * 2);
  auto ptr = thrust::device_ptr<SketchEntry const>(sketch_data.data());
  std::vector<SketchEntry> h_sketch_data(sketch_data.size());
  thrust::copy(ptr, ptr + sketch_data.size(), h_sketch_data.begin());

  for (size_t i = 0; i < kCols; ++i) {
    auto v_0 = h_sketch_data[i * 2];
    ASSERT_EQ(v_0.rmin, 0);
    ASSERT_EQ(v_0.wmin, kRows / 2.0f);
    ASSERT_EQ(v_0.rmax, kRows / 2.0f);

    auto v_1 = h_sketch_data[i * 2 + 1];
    ASSERT_EQ(v_1.rmin, kRows / 2.0f);
    ASSERT_EQ(v_1.wmin, kRows / 2.0f);
    ASSERT_EQ(v_1.rmax, static_cast<float>(kRows));
  }
}
}  // namespace common
}  // namespace xgboost
