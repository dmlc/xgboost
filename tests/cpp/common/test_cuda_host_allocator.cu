/**
 * Copyright 2024-2025, XGBoost Contributors
 */
#include <gtest/gtest.h>
#include <xgboost/context.h>  // for Context

#include <vector>

#include "../../../src/common/cuda_pinned_allocator.h"
#include "../../../src/common/device_helpers.cuh"  // for DefaultStream
#include "../../../src/common/numeric.h"           // for Iota

namespace xgboost {
TEST(CudaHostMalloc, Pinned) {
  std::vector<float, common::cuda_impl::PinnedAllocator<float>> vec;
  vec.resize(10);
  ASSERT_EQ(vec.size(), 10);
  Context ctx;
  common::Iota(&ctx, vec.begin(), vec.end(), 0);
  float k = 0;
  for (auto v : vec) {
    ASSERT_EQ(v, k);
    ++k;
  }
}

TEST(CudaHostMalloc, Managed) {
  std::vector<float, common::cuda_impl::ManagedAllocator<float>> vec;
  vec.resize(10);
#if defined(__linux__)
#if (CUDA_VERSION / 1000) >= 13
  hipMemLocation loc;
  loc.type = hipMemLocationTypeDevice;
  loc.id = 0;
  dh::safe_cuda(
      hipMemPrefetchAsync(vec.data(), vec.size() * sizeof(float), loc, 0, dh::DefaultStream()));
#else
  dh::safe_cuda(
      hipMemPrefetchAsync(vec.data(), vec.size() * sizeof(float), 0, dh::DefaultStream()));
#endif  // (CUDA_VERSION / 1000) >= 13
#endif
  dh::DefaultStream().Sync();
}
}  // namespace xgboost
