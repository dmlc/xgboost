/**
 * Copyright 2024, XGBoost Contributors
 */
#include <gtest/gtest.h>
#include <thread>  // for thread

#include <numeric>                     // for iota
#include <thrust/detail/sequence.inl>  // for sequence

#include "../../../src/common/cuda_rt_utils.h"     // for DrVersion
#include "../../../src/common/device_helpers.cuh"  // for CachingThrustPolicy, PinnedMemory
#include "../../../src/common/device_vector.cuh"
#include "xgboost/global_config.h"  // for GlobalConfigThreadLocalStore
#include "xgboost/windefs.h"  // for xgboost_IS_WIN

namespace dh {
TEST(DeviceUVector, Basic) {
  GlobalMemoryLogger().Clear();
  std::int32_t verbosity{3};
  std::swap(verbosity, xgboost::GlobalConfigThreadLocalStore::Get()->verbosity);
  DeviceUVector<float> uvec;
  uvec.resize(12);
  auto peak = GlobalMemoryLogger().PeakMemory();
  auto n_bytes = sizeof(decltype(uvec)::value_type) * uvec.size();
  ASSERT_EQ(peak, n_bytes);
  std::swap(verbosity, xgboost::GlobalConfigThreadLocalStore::Get()->verbosity);
}

#if defined(__linux__)
namespace {
class TestVirtualMem : public ::testing::TestWithParam<hipMemLocationType> {
 public:
  void Run() {
    auto type = this->GetParam();
    if (type == hipMemLocationTypeHostNuma) {
      GTEST_SKIP_("Host numa might require special system capabilities, skipping for now.");
    }
    detail::GrowOnlyVirtualMemVec vec{type};
    auto prop = xgboost::cudr::MakeAllocProp(type);
    auto gran = xgboost::cudr::GetAllocGranularity(&prop);
    ASSERT_GE(gran, 2);
    auto data = vec.GetSpan<std::int32_t>(32);  // should be smaller than granularity
    ASSERT_EQ(data.size(), 32);
    static_assert(std::is_same_v<typename decltype(data)::value_type, std::int32_t>);

    std::vector<std::int32_t> h_data(data.size());
    auto check = [&] {
      for (std::size_t i = 0; i < h_data.size(); ++i) {
        ASSERT_EQ(h_data[i], i);
      }
    };
    auto fill = [&](std::int32_t n_orig, xgboost::common::Span<std::int32_t> data) {
      if (type == hipMemLocationTypeDevice) {
        thrust::sequence(dh::CachingThrustPolicy(), data.data() + n_orig, data.data() + data.size(),
                         n_orig);
        dh::safe_cuda(hipMemcpy(h_data.data(), data.data(), data.size_bytes(), hipMemcpyDefault));
      } else {
        std::iota(data.data() + n_orig, data.data() + data.size(), n_orig);
        std::copy_n(data.data(), data.size(), h_data.data());
      }
    };

    fill(0, data);
    check();

    auto n_orig = data.size();
    // Should be smaller than granularity, use already reserved.
    data = vec.GetSpan<std::int32_t>(128);
    h_data.resize(data.size());
    fill(n_orig, data);
    check();
    if (128 < gran) {
      ASSERT_EQ(vec.Capacity(), gran);
    }

    n_orig = data.size();
    data = vec.GetSpan<std::int32_t>(gran / 2);
    h_data.resize(data.size());
    fill(n_orig, data);
    check();
    ASSERT_EQ(vec.Capacity(), gran * 2);

    n_orig = data.size();
    data = vec.GetSpan<std::int32_t>(gran);
    h_data.resize(data.size());
    fill(n_orig, data);
    check();
    ASSERT_EQ(vec.Capacity(), gran * 4);
  }
};
}  // anonymous namespace

TEST_P(TestVirtualMem, Alloc) { this->Run(); }

INSTANTIATE_TEST_SUITE_P(
    Basic, TestVirtualMem,
    ::testing::Values(hipMemLocationTypeDevice, hipMemLocationTypeHostNuma),
    [](::testing::TestParamInfo<TestVirtualMem::ParamType> const& info) -> char const* {
      auto type = info.param;
      switch (type) {
        case hipMemLocationTypeDevice:
          return "Device";
        case hipMemLocationTypeHostNuma:
          return "HostNuma";
        default:
          LOG(FATAL) << "unreachable";
      }
      return nullptr;
    });
#endif  // defined(__linux__)

TEST(TestVirtualMem, Version) {
  std::int32_t major, minor;
  xgboost::curt::DrVersion(&major, &minor);
  LOG(INFO) << "Latest supported CUDA version by the driver:" << major << "." << minor;
  PinnedMemory pinned;
  ASSERT_FALSE(pinned.IsVm());
}

TEST(AtomitFetch, Max) {
  auto n_threads = std::thread::hardware_concurrency();
  std::vector<std::thread> threads;
  std::atomic<std::int64_t> n{0};
  decltype(n)::value_type add = 64;
  for (decltype(n_threads) t = 0; t < n_threads; ++t) {
    threads.emplace_back([=, &n] {
      for (decltype(add) i = 0; i < add; ++i) {
        detail::AtomicFetchMax(n, static_cast<decltype(add)>(t + i));
      }
    });
  }
  for (auto& t : threads) {
    t.join();
  }
  ASSERT_EQ(n, n_threads - 1 + add - 1);  // 0-based indexing
}
}  // namespace dh
