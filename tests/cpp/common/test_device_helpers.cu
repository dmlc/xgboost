#include "hip/hip_runtime.h"

/*!
 * Copyright 2017 XGBoost contributors
 */
#include <thrust/device_vector.h>
#include <xgboost/base.h>
#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/quantile.h"
#include "../helpers.h"
#include "gtest/gtest.h"

TEST(SumReduce, Test) {
  thrust::device_vector<float> data(100, 1.0f);
  auto sum = dh::SumReduction(data.data().get(), data.size());
  ASSERT_NEAR(sum, 100.0f, 1e-5);
}

void TestAtomicSizeT() {
  size_t constexpr kThreads = 235;
  dh::device_vector<size_t> out(1, 0);
  auto d_out = dh::ToSpan(out);
  dh::LaunchN(0, kThreads, [=]__device__(size_t idx){
      atomicAdd(&d_out[0], static_cast<size_t>(1));
  });
  ASSERT_EQ(out[0], kThreads);
}

TEST(AtomicAdd, SizeT) {
  TestAtomicSizeT();
}

TEST(SegmentedUnique, Basic) {
  std::vector<float> values{0.1f, 0.2f, 0.3f, 0.62448811531066895f, 0.62448811531066895f, 0.4f};
  std::vector<size_t> segments{0, 3, 6};

  thrust::device_vector<float> d_values(values);
  thrust::device_vector<xgboost::bst_feature_t> d_segments{segments};

  thrust::device_vector<xgboost::bst_feature_t> d_segs_out(d_segments.size());
  thrust::device_vector<float> d_vals_out(d_values.size());

  size_t n_uniques = dh::SegmentedUnique(
      d_segments.data().get(), d_segments.data().get() + d_segments.size(),
      d_values.data().get(), d_values.data().get() + d_values.size(),
      d_segs_out.data().get(), d_vals_out.data().get(),
      thrust::equal_to<float>{});
  CHECK_EQ(n_uniques, 5);

  std::vector<float> values_sol{0.1f, 0.2f, 0.3f, 0.62448811531066895f, 0.4f};
  for (auto i = 0 ; i < values_sol.size(); i ++) {
    ASSERT_EQ(d_vals_out[i], values_sol[i]);
  }

  std::vector<xgboost::bst_feature_t> segments_sol{0, 3, 5};
  for (size_t i = 0; i < d_segments.size(); ++i) {
    ASSERT_EQ(segments_sol[i], d_segs_out[i]);
  }

  d_segments[1] = 4;
  d_segments[2] = 6;
  n_uniques = dh::SegmentedUnique(
      d_segments.data().get(), d_segments.data().get() + d_segments.size(),
      d_values.data().get(), d_values.data().get() + d_values.size(),
      d_segs_out.data().get(), d_vals_out.data().get(),
      thrust::equal_to<float>{});
  ASSERT_EQ(n_uniques, values.size());
  for (auto i = 0 ; i < values.size(); i ++) {
    ASSERT_EQ(d_vals_out[i], values[i]);
  }
}

namespace {
using SketchEntry = xgboost::common::WQSummary<float, float>::Entry;
struct SketchUnique {
  bool __device__ operator()(SketchEntry const& a, SketchEntry const& b) const {
    return a.value - b.value == 0;
  }
};
struct IsSorted {
  bool __device__ operator()(SketchEntry const& a, SketchEntry const& b) const {
    return a.value < b.value;
  }
};
}  // namespace

namespace xgboost {
namespace common {

void TestSegmentedUniqueRegression(std::vector<SketchEntry> values, size_t n_duplicated) {
  std::vector<bst_feature_t> segments{0, static_cast<bst_feature_t>(values.size())};

  thrust::device_vector<SketchEntry> d_values(values);
  thrust::device_vector<bst_feature_t> d_segments(segments);
  thrust::device_vector<bst_feature_t> d_segments_out(segments.size());

  size_t n_uniques = dh::SegmentedUnique(
      d_segments.data().get(), d_segments.data().get() + d_segments.size(), d_values.data().get(),
      d_values.data().get() + d_values.size(), d_segments_out.data().get(), d_values.data().get(),
      SketchUnique{});
  ASSERT_EQ(n_uniques, values.size() - n_duplicated);
  ASSERT_TRUE(thrust::is_sorted(thrust::device, d_values.begin(),
                                d_values.begin() + n_uniques, IsSorted{}));
  ASSERT_EQ(segments.at(0), d_segments_out[0]);
  ASSERT_EQ(segments.at(1), d_segments_out[1] + n_duplicated);
}


TEST(SegmentedUnique, Regression) {
  {
    std::vector<SketchEntry> values{{3149, 3150, 1, 0.62392902374267578},
                                    {3151, 3152, 1, 0.62418866157531738},
                                    {3152, 3153, 1, 0.62419462203979492},
                                    {3153, 3154, 1, 0.62431186437606812},
                                    {3154, 3155, 1, 0.6244881153106689453125},
                                    {3155, 3156, 1, 0.6244881153106689453125},
                                    {3155, 3156, 1, 0.6244881153106689453125},
                                    {3155, 3156, 1, 0.6244881153106689453125},
                                    {3157, 3158, 1, 0.62552797794342041},
                                    {3158, 3159, 1, 0.6256556510925293},
                                    {3159, 3160, 1, 0.62571090459823608},
                                    {3160, 3161, 1, 0.62577134370803833}};
    TestSegmentedUniqueRegression(values, 3);
  }
  {
    std::vector<SketchEntry> values{{3149, 3150, 1, 0.62392902374267578},
                                    {3151, 3152, 1, 0.62418866157531738},
                                    {3152, 3153, 1, 0.62419462203979492},
                                    {3153, 3154, 1, 0.62431186437606812},
                                    {3154, 3155, 1, 0.6244881153106689453125},
                                    {3157, 3158, 1, 0.62552797794342041},
                                    {3158, 3159, 1, 0.6256556510925293},
                                    {3159, 3160, 1, 0.62571090459823608},
                                    {3160, 3161, 1, 0.62577134370803833}};
    TestSegmentedUniqueRegression(values, 0);
  }
  {
    std::vector<SketchEntry> values;
    TestSegmentedUniqueRegression(values, 0);
  }
}
}  // namespace common
}  // namespace xgboost