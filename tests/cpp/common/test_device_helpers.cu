#include "hip/hip_runtime.h"
/*!
 * Copyright 2017 XGBoost contributors
 */
#include <thrust/device_vector.h>
#include <xgboost/base.h>
#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/quantile.h"
#include "../helpers.h"
#include "gtest/gtest.h"

TEST(SumReduce, Test) {
  thrust::device_vector<float> data(100, 1.0f);
  auto sum = dh::SumReduction(data.data().get(), data.size());
  ASSERT_NEAR(sum, 100.0f, 1e-5);
}

void TestAtomicSizeT() {
  size_t constexpr kThreads = 235;
  dh::device_vector<size_t> out(1, 0);
  auto d_out = dh::ToSpan(out);
  dh::LaunchN(0, kThreads, [=]__device__(size_t idx){
      atomicAdd(&d_out[0], static_cast<size_t>(1));
  });
  ASSERT_EQ(out[0], kThreads);
}

TEST(AtomicAdd, SizeT) {
  TestAtomicSizeT();
}

void TestSegmentID() {
  std::vector<size_t> segments{0, 1, 3};
  thrust::device_vector<size_t> d_segments(segments);
  auto s_segments = dh::ToSpan(d_segments);
  dh::LaunchN(0, 1, [=]__device__(size_t idx) {
    auto id = dh::SegmentId(s_segments, 0);
    SPAN_CHECK(id == 0);
    id = dh::SegmentId(s_segments, 1);
    SPAN_CHECK(id == 1);
    id = dh::SegmentId(s_segments, 2);
    SPAN_CHECK(id == 1);
  });
}

TEST(SegmentID, Basic) {
  TestSegmentID();
}

TEST(SegmentedUnique, Basic) {
  std::vector<float> values{0.1f, 0.2f, 0.3f, 0.62448811531066895f, 0.62448811531066895f, 0.4f};
  std::vector<size_t> segments{0, 3, 6};

  thrust::device_vector<float> d_values(values);
  thrust::device_vector<xgboost::bst_feature_t> d_segments{segments};

  thrust::device_vector<xgboost::bst_feature_t> d_segs_out(d_segments.size());
  thrust::device_vector<float> d_vals_out(d_values.size());

  size_t n_uniques = dh::SegmentedUnique(
      d_segments.data().get(), d_segments.data().get() + d_segments.size(),
      d_values.data().get(), d_values.data().get() + d_values.size(),
      d_segs_out.data().get(), d_vals_out.data().get(),
      thrust::equal_to<float>{});
  CHECK_EQ(n_uniques, 5);

  std::vector<float> values_sol{0.1f, 0.2f, 0.3f, 0.62448811531066895f, 0.4f};
  for (auto i = 0 ; i < values_sol.size(); i ++) {
    ASSERT_EQ(d_vals_out[i], values_sol[i]);
  }

  std::vector<xgboost::bst_feature_t> segments_sol{0, 3, 5};
  for (size_t i = 0; i < d_segments.size(); ++i) {
    ASSERT_EQ(segments_sol[i], d_segs_out[i]);
  }

  d_segments[1] = 4;
  d_segments[2] = 6;
  n_uniques = dh::SegmentedUnique(
      d_segments.data().get(), d_segments.data().get() + d_segments.size(),
      d_values.data().get(), d_values.data().get() + d_values.size(),
      d_segs_out.data().get(), d_vals_out.data().get(),
      thrust::equal_to<float>{});
  ASSERT_EQ(n_uniques, values.size());
  for (auto i = 0 ; i < values.size(); i ++) {
    ASSERT_EQ(d_vals_out[i], values[i]);
  }
}

namespace {
using SketchEntry = xgboost::common::WQSummary<float, float>::Entry;
struct SketchUnique {
  bool __device__ operator()(SketchEntry const& a, SketchEntry const& b) const {
    return a.value - b.value == 0;
  }
};
struct IsSorted {
  bool __device__ operator()(SketchEntry const& a, SketchEntry const& b) const {
    return a.value < b.value;
  }
};
}  // namespace

namespace xgboost {
namespace common {

void TestSegmentedUniqueRegression(std::vector<SketchEntry> values, size_t n_duplicated) {
  std::vector<bst_feature_t> segments{0, static_cast<bst_feature_t>(values.size())};

  thrust::device_vector<SketchEntry> d_values(values);
  thrust::device_vector<bst_feature_t> d_segments(segments);
  thrust::device_vector<bst_feature_t> d_segments_out(segments.size());

  size_t n_uniques = dh::SegmentedUnique(
      d_segments.data().get(), d_segments.data().get() + d_segments.size(), d_values.data().get(),
      d_values.data().get() + d_values.size(), d_segments_out.data().get(), d_values.data().get(),
      SketchUnique{});
  ASSERT_EQ(n_uniques, values.size() - n_duplicated);
  ASSERT_TRUE(thrust::is_sorted(thrust::device, d_values.begin(),
                                d_values.begin() + n_uniques, IsSorted{}));
  ASSERT_EQ(segments.at(0), d_segments_out[0]);
  ASSERT_EQ(segments.at(1), d_segments_out[1] + n_duplicated);
}

TEST(DeviceHelpers, Reduce) {
  size_t kSize = std::numeric_limits<uint32_t>::max();
  auto it = thrust::make_counting_iterator(0ul);
  dh::XGBCachingDeviceAllocator<char> alloc;
  auto batched = dh::Reduce(thrust::cuda::par(alloc), it, it + kSize, 0ul, thrust::maximum<size_t>{});
  CHECK_EQ(batched, kSize - 1);
}


TEST(SegmentedUnique, Regression) {
  {
    std::vector<SketchEntry> values{{3149, 3150, 1, 0.62392902374267578},
                                    {3151, 3152, 1, 0.62418866157531738},
                                    {3152, 3153, 1, 0.62419462203979492},
                                    {3153, 3154, 1, 0.62431186437606812},
                                    {3154, 3155, 1, 0.6244881153106689453125},
                                    {3155, 3156, 1, 0.6244881153106689453125},
                                    {3155, 3156, 1, 0.6244881153106689453125},
                                    {3155, 3156, 1, 0.6244881153106689453125},
                                    {3157, 3158, 1, 0.62552797794342041},
                                    {3158, 3159, 1, 0.6256556510925293},
                                    {3159, 3160, 1, 0.62571090459823608},
                                    {3160, 3161, 1, 0.62577134370803833}};
    TestSegmentedUniqueRegression(values, 3);
  }
  {
    std::vector<SketchEntry> values{{3149, 3150, 1, 0.62392902374267578},
                                    {3151, 3152, 1, 0.62418866157531738},
                                    {3152, 3153, 1, 0.62419462203979492},
                                    {3153, 3154, 1, 0.62431186437606812},
                                    {3154, 3155, 1, 0.6244881153106689453125},
                                    {3157, 3158, 1, 0.62552797794342041},
                                    {3158, 3159, 1, 0.6256556510925293},
                                    {3159, 3160, 1, 0.62571090459823608},
                                    {3160, 3161, 1, 0.62577134370803833}};
    TestSegmentedUniqueRegression(values, 0);
  }
  {
    std::vector<SketchEntry> values;
    TestSegmentedUniqueRegression(values, 0);
  }
}

TEST(Allocator, OOM) {
  auto size = dh::AvailableMemory(0) * 4;
  ASSERT_THROW({dh::caching_device_vector<char> vec(size);}, dmlc::Error);
  ASSERT_THROW({dh::device_vector<char> vec(size);}, dmlc::Error);
  // Clear last error so we don't fail subsequent tests
  hipGetLastError();
}
}  // namespace common
}  // namespace xgboost
