
/*!
 * Copyright 2017 XGBoost contributors
 */
#include <thrust/device_vector.h>
#include <xgboost/base.h>
#include "../../../src/common/device_helpers.cuh"
#include "../helpers.h"
#include "gtest/gtest.h"

using xgboost::common::Span;

void CreateTestData(xgboost::bst_uint num_rows, int max_row_size,
                    thrust::host_vector<int> *row_ptr,
                    thrust::host_vector<xgboost::bst_uint> *rows) {
  row_ptr->resize(num_rows + 1);
  int sum = 0;
  for (xgboost::bst_uint i = 0; i <= num_rows; i++) {
    (*row_ptr)[i] = sum;
    sum += rand() % max_row_size;  // NOLINT

    if (i < num_rows) {
      for (int j = (*row_ptr)[i]; j < sum; j++) {
        (*rows).push_back(i);
      }
    }
  }
}

void TestLbs() {
  srand(17);
  dh::CubMemory temp_memory;

  std::vector<int> test_rows = {4, 100, 1000};
  std::vector<int> test_max_row_sizes = {4, 100, 1300};

  for (auto num_rows : test_rows) {
    for (auto max_row_size : test_max_row_sizes) {
      thrust::host_vector<int> h_row_ptr;
      thrust::host_vector<xgboost::bst_uint> h_rows;
      CreateTestData(num_rows, max_row_size, &h_row_ptr, &h_rows);
      thrust::device_vector<size_t> row_ptr = h_row_ptr;
      thrust::device_vector<int> output_row(h_rows.size());
      auto d_output_row = output_row.data();

      dh::TransformLbs(0, &temp_memory, h_rows.size(), dh::Raw(row_ptr),
                       row_ptr.size() - 1, false,
                       [=] __device__(size_t idx, size_t ridx) {
                         d_output_row[idx] = ridx;
                       });

      dh::safe_cuda(hipDeviceSynchronize());
      ASSERT_TRUE(h_rows == output_row);
    }
  }
}

TEST(cub_lbs, Test) {
  TestLbs();
}

TEST(sumReduce, Test) {
  thrust::device_vector<float> data(100, 1.0f);
  dh::CubMemory temp;
  auto sum = dh::SumReduction(temp, dh::Raw(data), data.size());
  ASSERT_NEAR(sum, 100.0f, 1e-5);
}

void TestAllocator() {
  int n = 10;
  Span<float> a;
  Span<int> b;
  Span<size_t> c;
  dh::BulkAllocator ba;
  ba.Allocate(0, &a, n, &b, n, &c, n);

  // Should be no illegal memory accesses
  dh::LaunchN(0, n, [=] __device__(size_t idx) { c[idx] = a[idx] + b[idx]; });

  dh::safe_cuda(hipDeviceSynchronize());
}

// Define the test in a function so we can use device lambda
TEST(bulkAllocator, Test) {
  TestAllocator();
}

template <typename T, typename Comp = thrust::less<T>>
void TestUpperBoundImpl(const std::vector<T> &vec, T val_to_find,
                        const Comp &comp = Comp()) {
  EXPECT_EQ(dh::UpperBound(vec.data(), vec.size(), val_to_find, comp),
            std::upper_bound(vec.begin(), vec.end(), val_to_find, comp) - vec.begin());
}

template <typename T, typename Comp = thrust::less<T>>
void TestLowerBoundImpl(const std::vector<T> &vec, T val_to_find,
                        const Comp &comp = Comp()) {
  EXPECT_EQ(dh::LowerBound(vec.data(), vec.size(), val_to_find, comp),
            std::lower_bound(vec.begin(), vec.end(), val_to_find, comp) - vec.begin());
}

TEST(UpperBound, DataAscending) {
  std::vector<int> hvec{0, 3, 5, 5, 7, 8, 9, 10, 10};

  // Test boundary conditions
  TestUpperBoundImpl(hvec, hvec.front());  // Result 1
  TestUpperBoundImpl(hvec, hvec.front() - 1);  // Result 0
  TestUpperBoundImpl(hvec, hvec.back() + 1);  // Result hvec.size()
  TestUpperBoundImpl(hvec, hvec.back());  // Result hvec.size()

  // Test other values - both missing and present
  TestUpperBoundImpl(hvec, 3);  // Result 2
  TestUpperBoundImpl(hvec, 4);  // Result 2
  TestUpperBoundImpl(hvec, 5);  // Result 4
}

TEST(UpperBound, DataDescending) {
  std::vector<int> hvec{10, 10, 9, 8, 7, 5, 5, 3, 0, 0};
  const auto &comparator = thrust::greater<int>();

  // Test boundary conditions
  TestUpperBoundImpl(hvec, hvec.front(), comparator);  // Result 2
  TestUpperBoundImpl(hvec, hvec.front() + 1, comparator);  // Result 0
  TestUpperBoundImpl(hvec, hvec.back(), comparator);  // Result hvec.size()
  TestUpperBoundImpl(hvec, hvec.back() - 1, comparator);  // Result hvec.size()

  // Test other values - both missing and present
  TestUpperBoundImpl(hvec, 9, comparator);  // Result 3
  TestUpperBoundImpl(hvec, 7, comparator);  // Result 5
  TestUpperBoundImpl(hvec, 4, comparator);  // Result 7
  TestUpperBoundImpl(hvec, 8, comparator);  // Result 4
}

TEST(LowerBound, DataAscending) {
  std::vector<int> hvec{0, 3, 5, 5, 7, 8, 9, 10, 10};

  // Test boundary conditions
  TestLowerBoundImpl(hvec, hvec.front());  // Result 0
  TestLowerBoundImpl(hvec, hvec.front() - 1);  // Result 0
  TestLowerBoundImpl(hvec, hvec.back());  // Result 7
  TestLowerBoundImpl(hvec, hvec.back() + 1);  // Result hvec.size()

  // Test other values - both missing and present
  TestLowerBoundImpl(hvec, 3);  // Result 1
  TestLowerBoundImpl(hvec, 4);  // Result 2
  TestLowerBoundImpl(hvec, 5);  // Result 2
}

TEST(LowerBound, DataDescending) {
  std::vector<int> hvec{10, 10, 9, 8, 7, 5, 5, 3, 0, 0};
  const auto &comparator = thrust::greater<int>();

  // Test boundary conditions
  TestLowerBoundImpl(hvec, hvec.front(), comparator);  // Result 0
  TestLowerBoundImpl(hvec, hvec.front() + 1, comparator);  // Result 0
  TestLowerBoundImpl(hvec, hvec.back(), comparator);  // Result 8
  TestLowerBoundImpl(hvec, hvec.back() - 1, comparator);  // Result hvec.size()

  // Test other values - both missing and present
  TestLowerBoundImpl(hvec, 9, comparator);  // Result 2
  TestLowerBoundImpl(hvec, 7, comparator);  // Result 4
  TestLowerBoundImpl(hvec, 4, comparator);  // Result 7
  TestLowerBoundImpl(hvec, 8, comparator);  // Result 3
}
