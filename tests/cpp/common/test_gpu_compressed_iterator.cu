#include "../../../src/common/compressed_iterator.h"
#include "../../../src/common/device_helpers.cuh"
#include "gtest/gtest.h"
#include <algorithm>
#include <thrust/device_vector.h>

namespace xgboost {
namespace common {

struct WriteSymbolFunction {
  CompressedBufferWriter cbw;
  unsigned char* buffer_data_d;
  int* input_data_d;
  WriteSymbolFunction(CompressedBufferWriter cbw, unsigned char* buffer_data_d,
                      int* input_data_d)
    : cbw(cbw), buffer_data_d(buffer_data_d), input_data_d(input_data_d) {}

  __device__ void operator()(size_t i) {
    cbw.AtomicWriteSymbol(buffer_data_d, input_data_d[i], i);
  }
};

struct ReadSymbolFunction {
  CompressedIterator<int> ci;
  int* output_data_d;
  ReadSymbolFunction(CompressedIterator<int> ci, int* output_data_d)
    : ci(ci), output_data_d(output_data_d) {}

  __device__ void operator()(size_t i) {
    output_data_d[i] = ci[i];
  }
};

TEST(CompressedIterator, TestGPU) {
  dh::safe_cuda(hipSetDevice(0));
  std::vector<int> test_cases = {1, 3, 426, 21, 64, 256, 100000, INT32_MAX};
  int num_elements = 1000;
  int repetitions = 1000;
  srand(9);

  for (auto alphabet_size : test_cases) {
    for (int i = 0; i < repetitions; i++) {
      std::vector<int> input(num_elements);
      std::generate(input.begin(), input.end(),
        [=]() { return rand() % alphabet_size; });
      CompressedBufferWriter cbw(alphabet_size);
      thrust::device_vector<int> input_d(input);

      thrust::device_vector<unsigned char> buffer_d(
        CompressedBufferWriter::CalculateBufferSize(input.size(),
          alphabet_size));

      // write the data on device
      auto input_data_d = input_d.data().get();
      auto buffer_data_d = buffer_d.data().get();
      dh::LaunchN(input_d.size(),
                  WriteSymbolFunction(cbw, buffer_data_d, input_data_d));

      // read the data on device
      CompressedIterator<int> ci(buffer_d.data().get(), alphabet_size);
      thrust::device_vector<int> output_d(input.size());
      auto output_data_d = output_d.data().get();
      dh::LaunchN(output_d.size(), ReadSymbolFunction(ci, output_data_d));

      std::vector<int> output(output_d.size());
      thrust::copy(output_d.begin(), output_d.end(), output.begin());

      ASSERT_TRUE(input == output);
    }
  }
}

}  // namespace common
}  // namespace xgboost
