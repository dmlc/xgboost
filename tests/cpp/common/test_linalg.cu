/**
 * Copyright 2021-2025, XGBoost Contributors
 */
#include <gtest/gtest.h>
#include <thrust/equal.h>                       // for equal
#include <thrust/iterator/constant_iterator.h>  // for make_constant_iterator
#include <thrust/sequence.h>                    // for sequence

#include "../../../src/common/cuda_context.cuh"
#include "../../../src/common/linalg_op.cuh"
#include "../../../src/common/optional_weight.h"  // for MakeOptionalWeights
#include "../helpers.h"
#include "thrust/random.h"   // for default_random_engine
#include "thrust/shuffle.h"  // for shuffle
#include "xgboost/context.h"
#include "xgboost/linalg.h"

namespace xgboost::linalg {
namespace {
void TestElementWiseKernel() {
  auto device = DeviceOrd::CUDA(0);
  Tensor<float, 3> l{{2, 3, 4}, device};
  {
    /**
     * Non-contiguous
     */
    // GPU view
    auto t = l.View(device).Slice(linalg::All(), 1, linalg::All());
    ASSERT_FALSE(t.CContiguous());
    ElementWiseTransformDevice(t, [] __device__(size_t i, float) { return i; });
    // CPU view
    t = l.View(DeviceOrd::CPU()).Slice(linalg::All(), 1, linalg::All());
    std::size_t k = 0;
    for (size_t i = 0; i < l.Shape(0); ++i) {
      for (size_t j = 0; j < l.Shape(2); ++j) {
        ASSERT_EQ(k++, t(i, j));
      }
    }

    t = l.View(device).Slice(linalg::All(), 1, linalg::All());
    cuda_impl::ElementWiseKernel(
        t, [=] XGBOOST_DEVICE(std::size_t i, std::size_t j) mutable { t(i, j) = i + j; });

    t = l.Slice(linalg::All(), 1, linalg::All());
    for (size_t i = 0; i < l.Shape(0); ++i) {
      for (size_t j = 0; j < l.Shape(2); ++j) {
        ASSERT_EQ(i + j, t(i, j));
      }
    }
  }

  {
    /**
     * Contiguous
     */
    auto t = l.View(device);
    ElementWiseTransformDevice(t, [] XGBOOST_DEVICE(size_t i, float) { return i; });
    ASSERT_TRUE(t.CContiguous());
    // CPU view
    t = l.View(DeviceOrd::CPU());

    size_t ind = 0;
    for (size_t i = 0; i < l.Shape(0); ++i) {
      for (size_t j = 0; j < l.Shape(1); ++j) {
        for (size_t k = 0; k < l.Shape(2); ++k) {
          ASSERT_EQ(ind++, t(i, j, k));
        }
      }
    }
  }
}

void TestSlice() {
  auto ctx = MakeCUDACtx(1);
  thrust::device_vector<double> data(2 * 3 * 4);
  auto t = MakeTensorView(&ctx, dh::ToSpan(data), 2, 3, 4);
  dh::LaunchN(1, [=] __device__(size_t) {
    auto s = t.Slice(linalg::All(), linalg::Range(0, 3), linalg::Range(0, 4));
    auto all = t.Slice(linalg::All(), linalg::All(), linalg::All());
    static_assert(decltype(s)::kDimension == 3);
    for (size_t i = 0; i < s.Shape(0); ++i) {
      for (size_t j = 0; j < s.Shape(1); ++j) {
        for (size_t k = 0; k < s.Shape(2); ++k) {
          SPAN_CHECK(s(i, j, k) == all(i, j, k));
        }
      }
    }
  });
}

void TestWriteAccess(HIPContext const* cuctx, linalg::TensorView<double, 3> t) {
  thrust::for_each(cuctx->CTP(), linalg::tbegin(t), linalg::tend(t),
                   [=] XGBOOST_DEVICE(double& v) { v = 0; });
  auto eq = thrust::equal(cuctx->CTP(), linalg::tcbegin(t), linalg::tcend(t),
                          thrust::make_constant_iterator<double>(0.0), thrust::equal_to<>{});
  ASSERT_TRUE(eq);
}
}  // anonymous namespace

TEST(Linalg, GPUElementWise) { TestElementWiseKernel(); }

TEST(Linalg, GPUTensorView) { TestSlice(); }

TEST(Linalg, GPUIter) {
  auto ctx = MakeCUDACtx(1);
  auto cuctx = ctx.CUDACtx();

  dh::device_vector<double> data(2 * 3 * 4);
  thrust::sequence(cuctx->CTP(), data.begin(), data.end(), 1.0);

  auto t = MakeTensorView(&ctx, dh::ToSpan(data), 2, 3, 4);
  static_assert(!std::is_const_v<decltype(t)::element_type>);
  static_assert(!std::is_const_v<decltype(t)::value_type>);

  auto n = std::distance(linalg::tcbegin(t), linalg::tcend(t));
  ASSERT_EQ(n, t.Size());
  ASSERT_FALSE(t.Empty());

  bool eq = thrust::equal(cuctx->CTP(), data.cbegin(), data.cend(), linalg::tcbegin(t));
  ASSERT_TRUE(eq);

  TestWriteAccess(cuctx, t);
}

TEST(Linalg, SmallHistogram) {
  auto ctx = MakeCUDACtx(0);
  // Generate random data with 4 bins and 32 elements for each bin.
  std::size_t cnt = 32, n_bins = 4;
  dh::device_vector<float> values(cnt * n_bins);
  for (std::size_t i = 0; i < n_bins; ++i) {
    thrust::fill_n(ctx.CUDACtx()->CTP(), values.begin() + i * cnt, cnt, i);
  }
  thrust::default_random_engine rng;
  rng.seed(2025);
  thrust::shuffle(ctx.CUDACtx()->CTP(), values.begin(), values.end(), rng);

  linalg::MatrixView<float> indices =
      linalg::MakeTensorView(&ctx, dh::ToSpan(values), values.size(), 1);
  dh::CachingDeviceUVector<float> bins(n_bins);
  HostDeviceVector<float> weights;
  SmallHistogram(&ctx, indices, common::MakeOptionalWeights(&ctx, weights),
                 linalg::MakeTensorView(&ctx, dh::ToSpan(bins), bins.size()));

  std::vector<float> h_bins(n_bins);
  dh::safe_cuda(hipMemcpyAsync(h_bins.data(), bins.data(), dh::ToSpan(bins).size_bytes(),
                                hipMemcpyDefault, ctx.CUDACtx()->Stream()));
  for (std::size_t i = 0; i < n_bins; ++i) {
    ASSERT_EQ(h_bins[i], cnt);
  }
}
}  // namespace xgboost::linalg
