#include "hip/hip_runtime.h"
/*!
 * Copyright 2018 XGBoost contributors
 */
#include <gtest/gtest.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/span.h"
#include "test_span.h"

namespace xgboost {
namespace common {

struct TestStatus {
  int *status_;

 public:
  TestStatus () {
    dh::safe_cuda(hipMalloc(&status_, sizeof(int)));
    int h_status = 1;
    dh::safe_cuda(hipMemcpy(status_, &h_status,
                             sizeof(int), hipMemcpyHostToDevice));
  }
  ~TestStatus() {
    dh::safe_cuda(hipFree(status_));
  }

  int get() {
    int h_status;
    dh::safe_cuda(hipMemcpy(&h_status, status_,
                             sizeof(int), hipMemcpyDeviceToHost));
    return h_status;
  }

  int* data() {
    return status_;
  }
};

__global__ void test_from_other_kernel(Span<float> span) {
  // don't get optimized out
  size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= span.size())
    return;
}
// Test converting different T
  __global__ void test_from_other_kernel_const(Span<float const, 16> span) {
  // don't get optimized out
  size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= span.size())
    return;
}

/*!
 * \brief Here we just test whether the code compiles.
 */
TEST(GPUSpan, FromOther) {
  thrust::host_vector<float> h_vec (16);
  InitializeRange(h_vec.begin(), h_vec.end());

  thrust::device_vector<float> d_vec (h_vec.size());
  thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());
  // dynamic extent
  {
    Span<float> span (d_vec.data().get(), d_vec.size());
    test_from_other_kernel<<<1, 16>>>(span);
  }
  {
    Span<float> span (d_vec.data().get(), d_vec.size());
    test_from_other_kernel_const<<<1, 16>>>(span);
  }
  // static extent
  {
    Span<float, 16> span(d_vec.data().get(), d_vec.data().get() + 16);
    test_from_other_kernel<<<1, 16>>>(span);
  }
  {
    Span<float, 16> span(d_vec.data().get(), d_vec.data().get() + 16);
    test_from_other_kernel_const<<<1, 16>>>(span);
  }
}

TEST(GPUSpan, Assignment) {
  TestStatus status;
  dh::LaunchN(0, 16, TestAssignment{status.data()});
  ASSERT_EQ(status.get(), 1);
}

TEST(GPUSpan, TestStatus) {
  TestStatus status;
  dh::LaunchN(0, 16, TestTestStatus{status.data()});
  ASSERT_EQ(status.get(), -1);
}

template <typename T>
struct TestEqual {
  T *lhs_, *rhs_;
  int *status_;

  TestEqual(T* _lhs, T* _rhs, int * _status) :
      lhs_(_lhs), rhs_(_rhs), status_(_status) {}

  XGBOOST_DEVICE void operator()(size_t _idx) {
    bool res = lhs_[_idx] == rhs_[_idx];
    SPAN_ASSERT_TRUE(res, status_);
  }
};

TEST(GPUSpan, WithTrust) {
  // Not adviced to initialize span with host_vector, since h_vec.data() is
  // a host function.
  thrust::host_vector<float> h_vec (16);
  InitializeRange(h_vec.begin(), h_vec.end());

  thrust::device_vector<float> d_vec (h_vec.size());
  thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

  // Can't initialize span with device_vector, since d_vec.data() is not raw
  // pointer
  {
    Span<float> s (d_vec.data().get(), d_vec.size());

    ASSERT_EQ(d_vec.size(), s.size());
    ASSERT_EQ(d_vec.data().get(), s.data());
  }

  {
    TestStatus status;
    thrust::device_vector<float> d_vec1 (d_vec.size());
    thrust::copy(thrust::device, d_vec.begin(), d_vec.end(), d_vec1.begin());
    Span<float> s (d_vec1.data().get(), d_vec.size());

    dh::LaunchN(0, 16, TestEqual<float>{
        thrust::raw_pointer_cast(d_vec1.data()),
        s.data(), status.data()});
    ASSERT_EQ(status.get(), 1);

    // FIXME: memory error!
    // bool res = thrust::equal(thrust::device,
    //                          d_vec.begin(), d_vec.end(),
    //                          s.begin());
  }
}

TEST(GPUSpan, BeginEnd) {
  TestStatus status;
  dh::LaunchN(0, 16, TestBeginEnd{status.data()});
  ASSERT_EQ(status.get(), 1);
}

TEST(GPUSpan, RBeginREnd) {
  TestStatus status;
  dh::LaunchN(0, 16, TestRBeginREnd{status.data()});
  ASSERT_EQ(status.get(), 1);
}

__global__ void test_modify_kernel(Span<float> span) {
  size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= span.size())
    return;

  span[idx] = span.size() - idx;
}

TEST(GPUSpan, Modify) {
  thrust::host_vector<float> h_vec (16);
  InitializeRange(h_vec.begin(), h_vec.end());

  thrust::device_vector<float> d_vec (h_vec.size());
  thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

  Span<float> span (d_vec.data().get(), d_vec.size());

  test_modify_kernel<<<1, 16>>>(span);

  for (size_t i = 0; i < d_vec.size(); ++i) {
    ASSERT_EQ(d_vec[i], d_vec.size() - i);
  }
}

TEST(GPUSpan, Observers) {
  TestStatus status;
  dh::LaunchN(0, 16, TestObservers{status.data()});
  ASSERT_EQ(status.get(), 1);
}

TEST(GPUSpan, Compare) {
  TestStatus status;
  dh::LaunchN(0, 16, TestIterCompare{status.data()});
  ASSERT_EQ(status.get(), 1);
}

struct TestElementAccess {
  Span<float> span_;

  XGBOOST_DEVICE TestElementAccess (Span<float> _span) : span_(_span) {}

  XGBOOST_DEVICE float operator()(size_t _idx) {
    float tmp = span_[_idx];
    return tmp;
  }
};

TEST(GPUSpan, ElementAccess) {
  EXPECT_DEATH({
      thrust::host_vector<float> h_vec (16);
      InitializeRange(h_vec.begin(), h_vec.end());

      thrust::device_vector<float> d_vec (h_vec.size());
      thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

      Span<float> span (d_vec.data().get(), d_vec.size());
      dh::LaunchN(0, 17, TestElementAccess{span});}, "");
}

__global__ void test_first_dynamic_kernel(Span<float> _span) {
  _span.first<-1>();
}
__global__ void test_first_static_kernel(Span<float> _span) {
  _span.first(-1);
}
__global__ void test_last_dynamic_kernel(Span<float> _span) {
  _span.last<-1>();
}
__global__ void test_last_static_kernel(Span<float> _span) {
  _span.last(-1);
}

TEST(GPUSpan, FirstLast) {
  // We construct vectors multiple times since thrust can not recover from
  // death test.
  auto lambda_first_dy = []() {
    thrust::host_vector<float> h_vec (4);
    InitializeRange(h_vec.begin(), h_vec.end());

    thrust::device_vector<float> d_vec (h_vec.size());
    thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

    Span<float> span (d_vec.data().get(), d_vec.size());
    test_first_dynamic_kernel<<<1, 1>>>(span);
  };
  EXPECT_DEATH(lambda_first_dy(), "");

  auto lambda_first_static = []() {
    thrust::host_vector<float> h_vec (4);
    InitializeRange(h_vec.begin(), h_vec.end());

    thrust::device_vector<float> d_vec (h_vec.size());
    thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

    Span<float> span (d_vec.data().get(), d_vec.size());
    test_first_static_kernel<<<1, 1>>>(span);
  };
  EXPECT_DEATH(lambda_first_static(), "");

  auto lambda_last_dy = []() {
    thrust::host_vector<float> h_vec (4);
    InitializeRange(h_vec.begin(), h_vec.end());

    thrust::device_vector<float> d_vec (h_vec.size());
    thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

    Span<float> span (d_vec.data().get(), d_vec.size());
    test_last_dynamic_kernel<<<1, 1>>>(span);
  };
  EXPECT_DEATH(lambda_last_dy(), "");

  auto lambda_last_static = []() {
    thrust::host_vector<float> h_vec (4);
    InitializeRange(h_vec.begin(), h_vec.end());

    thrust::device_vector<float> d_vec (h_vec.size());
    thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

    Span<float> span (d_vec.data().get(), d_vec.size());
    test_last_static_kernel<<<1, 1>>>(span);
  };
  EXPECT_DEATH(lambda_last_static(), "");
}


__global__ void test_subspan_dynamic_kernel(Span<float> _span) {
  _span.subspan(16, 0);
}
__global__ void test_subspan_static_kernel(Span<float> _span) {
  _span.subspan<16>();
}
TEST(GPUSpan, Subspan) {
  auto lambda_subspan_dynamic = []() {
    thrust::host_vector<float> h_vec (4);
    InitializeRange(h_vec.begin(), h_vec.end());

    thrust::device_vector<float> d_vec (h_vec.size());
    thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

    Span<float> span (d_vec.data().get(), d_vec.size());
    test_subspan_dynamic_kernel<<<1, 1>>>(span);
  };
  EXPECT_DEATH(lambda_subspan_dynamic(), "");

  auto lambda_subspan_static = []() {
    thrust::host_vector<float> h_vec (4);
    InitializeRange(h_vec.begin(), h_vec.end());

    thrust::device_vector<float> d_vec (h_vec.size());
    thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());

    Span<float> span (d_vec.data().get(), d_vec.size());
    test_subspan_static_kernel<<<1, 1>>>(span);
  };
  EXPECT_DEATH(lambda_subspan_static(), "");
}

TEST(GPUSpanIter, Construct) {
  TestStatus status;
  dh::LaunchN(0, 16, TestIterConstruct{status.data()});
  ASSERT_EQ(status.get(), 1);
}

TEST(GPUSpanIter, Ref) {
  TestStatus status;
  dh::LaunchN(0, 16, TestIterRef{status.data()});
  ASSERT_EQ(status.get(), 1);
}

TEST(GPUSpanIter, Calculate) {
  TestStatus status;
  dh::LaunchN(0, 16, TestIterCalculate{status.data()});
  ASSERT_EQ(status.get(), 1);
}

TEST(GPUSpanIter, Compare) {
  TestStatus status;
  dh::LaunchN(0, 16, TestIterCompare{status.data()});
  ASSERT_EQ(status.get(), 1);
}

TEST(GPUSpan, AsBytes) {
  TestStatus status;
  dh::LaunchN(0, 16, TestAsBytes{status.data()});
  ASSERT_EQ(status.get(), 1);
}

TEST(GPUSpan, AsWritableBytes) {
  TestStatus status;
  dh::LaunchN(0, 16, TestAsWritableBytes{status.data()});
  ASSERT_EQ(status.get(), 1);
}

}  // namespace common
}  // namespace xgboost
