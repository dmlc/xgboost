/**
 * Copyright 2024, XGBoost Contributors
 */
#if defined(__linux__)

#include <gtest/gtest.h>
#include <thrust/equal.h>                       // for equal
#include <thrust/fill.h>                        // for fill_n
#include <thrust/iterator/constant_iterator.h>  // for make_constant_iterator
#include <thrust/sequence.h>                    // for sequence

#include "../../../src/common/ref_resource_view.cuh"
#include "../helpers.h"  // for MakeCUDACtx

namespace xgboost::common {
class TestCudaGrowOnly : public ::testing::TestWithParam<std::size_t> {
 public:
  void TestGrow(std::size_t m, std::size_t n) {
    auto ctx = MakeCUDACtx(0);
    ctx.CUDACtx()->Stream().Sync();

    auto ref = MakeCudaGrowOnly<double>(m);
    ASSERT_EQ(ref.size_bytes(), m * sizeof(double));
    thrust::sequence(ctx.CUDACtx()->CTP(), ref.begin(), ref.end(), 0.0);
    auto res = std::dynamic_pointer_cast<common::CudaGrowOnlyResource>(ref.Resource());
    CHECK(res);
    res->Resize(n * sizeof(double));

    auto ref1 = RefResourceView<double>(res->DataAs<double>(), res->Size() / sizeof(double),
                                        ref.Resource());
    ASSERT_EQ(res->Size(), n * sizeof(double));
    ASSERT_EQ(ref1.size(), n);
    thrust::sequence(ctx.CUDACtx()->CTP(), ref1.begin(), ref1.end(), static_cast<double>(0.0));
    std::vector<double> h_vec(ref1.size());
    dh::safe_cuda(hipMemcpyAsync(h_vec.data(), ref1.data(), ref1.size_bytes(), hipMemcpyDefault));
    for (std::size_t i = 0; i < h_vec.size(); ++i) {
      ASSERT_EQ(h_vec[i], i);
    }
  }

  void Run(std::size_t n) { this->TestGrow(1024, n); }
};

TEST_P(TestCudaGrowOnly, Resize) { this->Run(this->GetParam()); }

INSTANTIATE_TEST_SUITE_P(RefResourceView, TestCudaGrowOnly, ::testing::Values(1 << 20, 1 << 21));
}  // namespace xgboost::common

#endif  // defined(__linux__)
