#include "hip/hip_runtime.h"
/**
 * Copyright 2019-2023 by XGBoost Contributors
 */
#include <gtest/gtest.h>
#include <thrust/device_vector.h>
#include <xgboost/base.h>  // for bst_bin_t
#include <xgboost/c_api.h>
#include <xgboost/data.h>

#include <algorithm>  // for transform
#include <cmath>      // for floor
#include <cstddef>    // for size_t
#include <limits>     // for numeric_limits
#include <string>     // for string, to_string
#include <tuple>      // for tuple, make_tuple
#include <vector>     // for vector

#include "../../../include/xgboost/logging.h"
#include "../../../src/common/cuda_context.cuh"
#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/hist_util.cuh"
#include "../../../src/common/hist_util.h"
#include "../../../src/data/device_adapter.cuh"
#include "../../../src/data/simple_dmatrix.h"
#include "../data/test_array_interface.h"
#include "../filesystem.h"  // dmlc::TemporaryDirectory
#include "../helpers.h"
#include "test_hist_util.h"

namespace xgboost::common {

template <typename AdapterT>
HistogramCuts GetHostCuts(Context const* ctx, AdapterT* adapter, int num_bins, float missing) {
  data::SimpleDMatrix dmat(adapter, missing, 1);
  HistogramCuts cuts = SketchOnDMatrix(ctx, &dmat, num_bins);
  return cuts;
}

TEST(HistUtil, DeviceSketch) {
  auto ctx = MakeCUDACtx(0);
  int num_columns = 1;
  int num_bins = 4;
  std::vector<float> x = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 7.0f, -1.0f};
  int num_rows = x.size();
  auto dmat = GetDMatrixFromData(x, num_rows, num_columns);

  auto device_cuts = DeviceSketch(&ctx, dmat.get(), num_bins);

  Context cpu_ctx;
  HistogramCuts host_cuts = SketchOnDMatrix(&cpu_ctx, dmat.get(), num_bins);

  EXPECT_EQ(device_cuts.Values(), host_cuts.Values());
  EXPECT_EQ(device_cuts.Ptrs(), host_cuts.Ptrs());
  EXPECT_EQ(device_cuts.MinValues(), host_cuts.MinValues());
}

TEST(HistUtil, SketchBatchNumElements) {
#if defined(XGBOOST_USE_RMM) && XGBOOST_USE_RMM == 1
  LOG(WARNING) << "Test not runnable with RMM enabled.";
  return;
#endif  // defined(XGBOOST_USE_RMM) && XGBOOST_USE_RMM == 1
  size_t constexpr kCols = 10000;
  int device;
  dh::safe_cuda(hipGetDevice(&device));
  auto avail = static_cast<size_t>(dh::AvailableMemory(device) * 0.8);
  auto per_elem = detail::BytesPerElement(false);
  auto avail_elem = avail / per_elem;
  size_t rows = avail_elem / kCols * 10;
  auto batch = detail::SketchBatchNumElements(0, rows, kCols, rows * kCols, device, 256, false);
  ASSERT_EQ(batch, avail_elem);
}

TEST(HistUtil, DeviceSketchMemory) {
  auto ctx = MakeCUDACtx(0);
  int num_columns = 100;
  int num_rows = 1000;
  int num_bins = 256;
  auto x = GenerateRandom(num_rows, num_columns);
  auto dmat = GetDMatrixFromData(x, num_rows, num_columns);

  dh::GlobalMemoryLogger().Clear();
  ConsoleLogger::Configure({{"verbosity", "3"}});
  auto device_cuts = DeviceSketch(&ctx, dmat.get(), num_bins);

  size_t bytes_required = detail::RequiredMemory(
      num_rows, num_columns, num_rows * num_columns, num_bins, false);
  EXPECT_LE(dh::GlobalMemoryLogger().PeakMemory(), bytes_required * 1.05);
  EXPECT_GE(dh::GlobalMemoryLogger().PeakMemory(), bytes_required * 0.95);
  ConsoleLogger::Configure({{"verbosity", "0"}});
}

TEST(HistUtil, DeviceSketchWeightsMemory) {
  auto ctx = MakeCUDACtx(0);
  int num_columns = 100;
  int num_rows = 1000;
  int num_bins = 256;
  auto x = GenerateRandom(num_rows, num_columns);
  auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
  dmat->Info().weights_.HostVector() = GenerateRandomWeights(num_rows);

  dh::GlobalMemoryLogger().Clear();
  ConsoleLogger::Configure({{"verbosity", "3"}});
  auto device_cuts = DeviceSketch(&ctx, dmat.get(), num_bins);
  ConsoleLogger::Configure({{"verbosity", "0"}});

  size_t bytes_required = detail::RequiredMemory(
      num_rows, num_columns, num_rows * num_columns, num_bins, true);
  EXPECT_LE(dh::GlobalMemoryLogger().PeakMemory(), bytes_required * 1.05);
  EXPECT_GE(dh::GlobalMemoryLogger().PeakMemory(), bytes_required);
}

TEST(HistUtil, DeviceSketchDeterminism) {
  auto ctx = MakeCUDACtx(0);
  int num_rows = 500;
  int num_columns = 5;
  int num_bins = 256;
  auto x = GenerateRandom(num_rows, num_columns);
  auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
  auto reference_sketch = DeviceSketch(&ctx, dmat.get(), num_bins);
  size_t constexpr kRounds{ 100 };
  for (size_t r = 0; r < kRounds; ++r) {
    auto new_sketch = DeviceSketch(&ctx, dmat.get(), num_bins);
    ASSERT_EQ(reference_sketch.Values(), new_sketch.Values());
    ASSERT_EQ(reference_sketch.MinValues(), new_sketch.MinValues());
  }
}

TEST(HistUtil, DeviceSketchCategoricalAsNumeric) {
  auto ctx = MakeCUDACtx(0);
  auto categorical_sizes = {2, 6, 8, 12};
  int num_bins = 256;
  auto sizes = {25, 100, 1000};
  for (auto n : sizes) {
    for (auto num_categories : categorical_sizes) {
      auto x = GenerateRandomCategoricalSingleColumn(n, num_categories);
      auto dmat = GetDMatrixFromData(x, n, 1);
      auto cuts = DeviceSketch(&ctx, dmat.get(), num_bins);
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}

TEST(HistUtil, DeviceSketchCategoricalFeatures) {
  auto ctx = MakeCUDACtx(0);
  TestCategoricalSketch(1000, 256, 32, false, [ctx](DMatrix* p_fmat, int32_t num_bins) {
    return DeviceSketch(&ctx, p_fmat, num_bins);
  });
  TestCategoricalSketch(1000, 256, 32, true, [ctx](DMatrix* p_fmat, int32_t num_bins) {
    return DeviceSketch(&ctx, p_fmat, num_bins);
  });
}

void TestMixedSketch() {
  size_t n_samples = 1000, n_features = 2, n_categories = 3;
  bst_bin_t n_bins = 64;

  std::vector<float> data(n_samples * n_features);
  SimpleLCG gen;
  SimpleRealUniformDistribution<float> cat_d{0.0f, static_cast<float>(n_categories)};
  SimpleRealUniformDistribution<float> num_d{0.0f, 3.0f};
  for (size_t i = 0; i < n_samples * n_features; ++i) {
    // two features, row major. The first column is numeric and the second is categorical.
    if (i % 2 == 0) {
      data[i] = std::floor(cat_d(&gen));
    } else {
      data[i] = num_d(&gen);
    }
  }

  auto m = GetDMatrixFromData(data, n_samples, n_features);
  m->Info().feature_types.HostVector().push_back(FeatureType::kCategorical);
  m->Info().feature_types.HostVector().push_back(FeatureType::kNumerical);

  auto ctx = MakeCUDACtx(0);
  auto cuts = DeviceSketch(&ctx, m.get(), n_bins);
  ASSERT_EQ(cuts.Values().size(), n_bins + n_categories);
}

TEST(HistUtil, DeviceSketchMixedFeatures) { TestMixedSketch(); }

TEST(HistUtil, RemoveDuplicatedCategories) {
  bst_row_t n_samples = 512;
  bst_feature_t n_features = 3;
  bst_cat_t n_categories = 5;

  auto ctx = MakeCUDACtx(0);
  SimpleLCG rng;
  SimpleRealUniformDistribution<float> cat_d{0.0f, static_cast<float>(n_categories)};

  dh::device_vector<Entry> sorted_entries(n_samples * n_features);
  for (std::size_t i = 0; i < n_samples; ++i) {
    for (bst_feature_t j = 0; j < n_features; ++j) {
      float fvalue{0.0f};
      // The second column is categorical
      if (j == 1) {
        fvalue = std::floor(cat_d(&rng));
      } else {
        fvalue = i;
      }
      sorted_entries[i * n_features + j] = Entry{j, fvalue};
    }
  }

  MetaInfo info;
  info.num_col_ = n_features;
  info.num_row_ = n_samples;
  info.feature_types.HostVector() = std::vector<FeatureType>{
      FeatureType::kNumerical, FeatureType::kCategorical, FeatureType::kNumerical};
  ASSERT_EQ(info.feature_types.Size(), n_features);

  HostDeviceVector<bst_row_t> cuts_ptr{0, n_samples, n_samples * 2, n_samples * 3};
  cuts_ptr.SetDevice(DeviceOrd::CUDA(0));

  dh::device_vector<float> weight(n_samples * n_features, 0);
  dh::Iota(dh::ToSpan(weight), ctx.CUDACtx()->Stream());

  dh::caching_device_vector<bst_row_t> columns_ptr(4);
  for (std::size_t i = 0; i < columns_ptr.size(); ++i) {
    columns_ptr[i] = i * n_samples;
  }
  // sort into column major
  thrust::sort_by_key(sorted_entries.begin(), sorted_entries.end(), weight.begin(),
                      detail::EntryCompareOp());

  detail::RemoveDuplicatedCategories(ctx.Device(), info, cuts_ptr.DeviceSpan(), &sorted_entries,
                                     &weight, &columns_ptr);

  auto const& h_cptr = cuts_ptr.ConstHostVector();
  ASSERT_EQ(h_cptr.back(), n_samples * 2 + n_categories);
  // check numerical
  for (std::size_t i = 0; i < n_samples; ++i) {
    ASSERT_EQ(weight[i], i * 3);
  }
  auto beg = n_samples + n_categories;
  for (std::size_t i = 0; i < n_samples; ++i) {
    ASSERT_EQ(weight[i + beg], i * 3 + 2);
  }
  // check categorical
  beg = n_samples;
  for (bst_cat_t i = 0; i < n_categories; ++i) {
    // all from the second column
    ASSERT_EQ(static_cast<bst_feature_t>(weight[i + beg]) % n_features, 1);
  }
}

TEST(HistUtil, DeviceSketchMultipleColumns) {
  auto ctx = MakeCUDACtx(0);
  auto bin_sizes = {2, 16, 256, 512};
  auto sizes = {100, 1000, 1500};
  int num_columns = 5;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    for (auto num_bins : bin_sizes) {
      auto cuts = DeviceSketch(&ctx, dmat.get(), num_bins);
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}

TEST(HistUtil, DeviceSketchMultipleColumnsWeights) {
  auto ctx = MakeCUDACtx(0);
  auto bin_sizes = {2, 16, 256, 512};
  auto sizes = {100, 1000, 1500};
  int num_columns = 5;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    dmat->Info().weights_.HostVector() = GenerateRandomWeights(num_rows);
    for (auto num_bins : bin_sizes) {
      auto cuts = DeviceSketch(&ctx, dmat.get(), num_bins);
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}

TEST(HistUitl, DeviceSketchWeights) {
  auto ctx = MakeCUDACtx(0);
  auto bin_sizes = {2, 16, 256, 512};
  auto sizes = {100, 1000, 1500};
  int num_columns = 5;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    auto weighted_dmat = GetDMatrixFromData(x, num_rows, num_columns);
    auto& h_weights = weighted_dmat->Info().weights_.HostVector();
    h_weights.resize(num_rows);
    std::fill(h_weights.begin(), h_weights.end(), 1.0f);
    for (auto num_bins : bin_sizes) {
      auto cuts = DeviceSketch(&ctx, dmat.get(), num_bins);
      auto wcuts = DeviceSketch(&ctx, weighted_dmat.get(), num_bins);
      ASSERT_EQ(cuts.MinValues(), wcuts.MinValues());
      ASSERT_EQ(cuts.Ptrs(), wcuts.Ptrs());
      ASSERT_EQ(cuts.Values(), wcuts.Values());
      ValidateCuts(cuts, dmat.get(), num_bins);
      ValidateCuts(wcuts, weighted_dmat.get(), num_bins);
    }
  }
}

TEST(HistUtil, DeviceSketchBatches) {
  auto ctx = MakeCUDACtx(0);
  int num_bins = 256;
  int num_rows = 5000;
  auto batch_sizes = {0, 100, 1500, 6000};
  int num_columns = 5;
  for (auto batch_size : batch_sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    auto cuts = DeviceSketch(&ctx, dmat.get(), num_bins, batch_size);
    ValidateCuts(cuts, dmat.get(), num_bins);
  }

  num_rows = 1000;
  size_t batches = 16;
  auto x = GenerateRandom(num_rows * batches, num_columns);
  auto dmat = GetDMatrixFromData(x, num_rows * batches, num_columns);
  auto cuts_with_batches = DeviceSketch(&ctx, dmat.get(), num_bins, num_rows);
  auto cuts = DeviceSketch(&ctx, dmat.get(), num_bins, 0);

  auto const& cut_values_batched = cuts_with_batches.Values();
  auto const& cut_values = cuts.Values();
  CHECK_EQ(cut_values.size(), cut_values_batched.size());
  for (size_t i = 0; i < cut_values.size(); ++i) {
    ASSERT_NEAR(cut_values_batched[i], cut_values[i], 1e5);
  }
}

TEST(HistUtil, DeviceSketchMultipleColumnsExternal) {
  auto ctx = MakeCUDACtx(0);
  auto bin_sizes = {2, 16, 256, 512};
  auto sizes = {100, 1000, 1500};
  int num_columns =5;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    dmlc::TemporaryDirectory temp;
    auto dmat = GetExternalMemoryDMatrixFromData(x, num_rows, num_columns, temp);
    for (auto num_bins : bin_sizes) {
      auto cuts = DeviceSketch(&ctx, dmat.get(), num_bins);
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}

// See https://github.com/dmlc/xgboost/issues/5866.
TEST(HistUtil, DeviceSketchExternalMemoryWithWeights) {
  auto ctx = MakeCUDACtx(0);
  auto bin_sizes = {2, 16, 256, 512};
  auto sizes = {100, 1000, 1500};
  int num_columns = 5;
  dmlc::TemporaryDirectory temp;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetExternalMemoryDMatrixFromData(x, num_rows, num_columns, temp);
    dmat->Info().weights_.HostVector() = GenerateRandomWeights(num_rows);
    for (auto num_bins : bin_sizes) {
      auto cuts = DeviceSketch(&ctx, dmat.get(), num_bins);
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}

template <typename Adapter>
auto MakeUnweightedCutsForTest(Context const* ctx, Adapter adapter, int32_t num_bins, float missing,
                               size_t batch_size = 0) {
  common::HistogramCuts batched_cuts;
  HostDeviceVector<FeatureType> ft;
  SketchContainer sketch_container(ft, num_bins, adapter.NumColumns(), adapter.NumRows(),
                                   DeviceOrd::CUDA(0));
  MetaInfo info;
  AdapterDeviceSketch(adapter.Value(), num_bins, info, missing, &sketch_container, batch_size);
  sketch_container.MakeCuts(ctx, &batched_cuts, info.IsColumnSplit());
  return batched_cuts;
}

template <typename Adapter>
void ValidateBatchedCuts(Context const* ctx, Adapter adapter, int num_bins, DMatrix* dmat, size_t batch_size = 0) {
  common::HistogramCuts batched_cuts = MakeUnweightedCutsForTest(
      ctx, adapter, num_bins, std::numeric_limits<float>::quiet_NaN(), batch_size);
  ValidateCuts(batched_cuts, dmat, num_bins);
}

TEST(HistUtil, AdapterDeviceSketch) {
  auto ctx = MakeCUDACtx(0);
  int rows = 5;
  int cols = 1;
  int num_bins = 4;
  float missing =  - 1.0;
  thrust::device_vector< float> data(rows*cols);
  auto json_array_interface = Generate2dArrayInterface(rows, cols, "<f4", &data);
  data = std::vector<float >{ 1.0,2.0,3.0,4.0,5.0 };
  std::string str;
  Json::Dump(json_array_interface, &str);

  data::CupyAdapter adapter(str);

  auto device_cuts = MakeUnweightedCutsForTest(&ctx, adapter, num_bins, missing);
  ctx = ctx.MakeCPU();
  auto host_cuts = GetHostCuts(&ctx, &adapter, num_bins, missing);

  EXPECT_EQ(device_cuts.Values(), host_cuts.Values());
  EXPECT_EQ(device_cuts.Ptrs(), host_cuts.Ptrs());
  EXPECT_EQ(device_cuts.MinValues(), host_cuts.MinValues());
}

TEST(HistUtil, AdapterDeviceSketchMemory) {
  auto ctx = MakeCUDACtx(0);
  int num_columns = 100;
  int num_rows = 1000;
  int num_bins = 256;
  auto x = GenerateRandom(num_rows, num_columns);
  auto x_device = thrust::device_vector<float>(x);
  auto adapter = AdapterFromData(x_device, num_rows, num_columns);

  dh::GlobalMemoryLogger().Clear();
  ConsoleLogger::Configure({{"verbosity", "3"}});
  auto cuts =
      MakeUnweightedCutsForTest(&ctx, adapter, num_bins, std::numeric_limits<float>::quiet_NaN());
  ConsoleLogger::Configure({{"verbosity", "0"}});
  size_t bytes_required = detail::RequiredMemory(
      num_rows, num_columns, num_rows * num_columns, num_bins, false);
  EXPECT_LE(dh::GlobalMemoryLogger().PeakMemory(), bytes_required * 1.05);
  EXPECT_GE(dh::GlobalMemoryLogger().PeakMemory(), bytes_required * 0.95);
}

TEST(HistUtil, AdapterSketchSlidingWindowMemory) {
  auto ctx = MakeCUDACtx(0);
  int num_columns = 100;
  int num_rows = 1000;
  int num_bins = 256;
  auto x = GenerateRandom(num_rows, num_columns);
  auto x_device = thrust::device_vector<float>(x);
  auto adapter = AdapterFromData(x_device, num_rows, num_columns);
  MetaInfo info;

  dh::GlobalMemoryLogger().Clear();
  ConsoleLogger::Configure({{"verbosity", "3"}});
  common::HistogramCuts batched_cuts;
  HostDeviceVector<FeatureType> ft;
  SketchContainer sketch_container(ft, num_bins, num_columns, num_rows, DeviceOrd::CUDA(0));
  AdapterDeviceSketch(adapter.Value(), num_bins, info, std::numeric_limits<float>::quiet_NaN(),
                      &sketch_container);
  HistogramCuts cuts;
  sketch_container.MakeCuts(&ctx, &cuts, info.IsColumnSplit());
  size_t bytes_required = detail::RequiredMemory(
      num_rows, num_columns, num_rows * num_columns, num_bins, false);
  EXPECT_LE(dh::GlobalMemoryLogger().PeakMemory(), bytes_required * 1.05);
  EXPECT_GE(dh::GlobalMemoryLogger().PeakMemory(), bytes_required * 0.95);
  ConsoleLogger::Configure({{"verbosity", "0"}});
}

TEST(HistUtil, AdapterSketchSlidingWindowWeightedMemory) {
  auto ctx = MakeCUDACtx(0);
  int num_columns = 100;
  int num_rows = 1000;
  int num_bins = 256;
  auto x = GenerateRandom(num_rows, num_columns);
  auto x_device = thrust::device_vector<float>(x);
  auto adapter = AdapterFromData(x_device, num_rows, num_columns);
  MetaInfo info;
  auto& h_weights = info.weights_.HostVector();
  h_weights.resize(num_rows);
  std::fill(h_weights.begin(), h_weights.end(), 1.0f);

  dh::GlobalMemoryLogger().Clear();
  ConsoleLogger::Configure({{"verbosity", "3"}});
  common::HistogramCuts batched_cuts;
  HostDeviceVector<FeatureType> ft;
  SketchContainer sketch_container(ft, num_bins, num_columns, num_rows, DeviceOrd::CUDA(0));
  AdapterDeviceSketch(adapter.Value(), num_bins, info,
                      std::numeric_limits<float>::quiet_NaN(),
                      &sketch_container);

  HistogramCuts cuts;
  sketch_container.MakeCuts(&ctx, &cuts, info.IsColumnSplit());
  ConsoleLogger::Configure({{"verbosity", "0"}});
  size_t bytes_required = detail::RequiredMemory(
      num_rows, num_columns, num_rows * num_columns, num_bins, true);
  EXPECT_LE(dh::GlobalMemoryLogger().PeakMemory(), bytes_required * 1.05);
  EXPECT_GE(dh::GlobalMemoryLogger().PeakMemory(), bytes_required);
}

void TestCategoricalSketchAdapter(size_t n, size_t num_categories,
                                  int32_t num_bins, bool weighted) {
  auto ctx = MakeCUDACtx(0);
  auto h_x = GenerateRandomCategoricalSingleColumn(n, num_categories);
  thrust::device_vector<float> x(h_x);
  auto adapter = AdapterFromData(x, n, 1);
  MetaInfo info;
  info.num_row_ = n;
  info.num_col_ = 1;
  info.feature_types.HostVector().push_back(FeatureType::kCategorical);

  if (weighted) {
    std::vector<float> weights(n, 0);
    SimpleLCG lcg;
    SimpleRealUniformDistribution<float> dist(0, 1);
    for (auto& v : weights) {
      v = dist(&lcg);
    }
    info.weights_.HostVector() = weights;
  }

  ASSERT_EQ(info.feature_types.Size(), 1);
  SketchContainer container(info.feature_types, num_bins, 1, n, DeviceOrd::CUDA(0));
  AdapterDeviceSketch(adapter.Value(), num_bins, info,
                      std::numeric_limits<float>::quiet_NaN(), &container);
  HistogramCuts cuts;
  container.MakeCuts(&ctx, &cuts, info.IsColumnSplit());

  thrust::sort(x.begin(), x.end());
  auto n_uniques = thrust::unique(x.begin(), x.end()) - x.begin();
  ASSERT_NE(n_uniques, x.size());
  ASSERT_EQ(cuts.TotalBins(), n_uniques);
  ASSERT_EQ(n_uniques, num_categories);

  auto& values = cuts.cut_values_.HostVector();
  ASSERT_TRUE(std::is_sorted(values.cbegin(), values.cend()));
  auto is_unique = (std::unique(values.begin(), values.end()) - values.begin()) == n_uniques;
  ASSERT_TRUE(is_unique);

  x.resize(n_uniques);
  h_x.resize(n_uniques);
  thrust::copy(x.begin(), x.end(), h_x.begin());
  for (decltype(n_uniques) i = 0; i < n_uniques; ++i) {
    ASSERT_EQ(h_x[i], values[i]);
  }
}

TEST(HistUtil, AdapterDeviceSketchCategorical) {
  auto categorical_sizes = {2, 6, 8, 12};
  int num_bins = 256;
  auto ctx = MakeCUDACtx(0);
  auto sizes = {25, 100, 1000};
  for (auto n : sizes) {
    for (auto num_categories : categorical_sizes) {
      auto x = GenerateRandomCategoricalSingleColumn(n, num_categories);
      auto dmat = GetDMatrixFromData(x, n, 1);
      auto x_device = thrust::device_vector<float>(x);
      auto adapter = AdapterFromData(x_device, n, 1);
      ValidateBatchedCuts(&ctx, adapter, num_bins, dmat.get());
      TestCategoricalSketchAdapter(n, num_categories, num_bins, true);
      TestCategoricalSketchAdapter(n, num_categories, num_bins, false);
    }
  }
}

TEST(HistUtil, AdapterDeviceSketchMultipleColumns) {
  auto bin_sizes = {2, 16, 256, 512};
  auto sizes = {100, 1000, 1500};
  int num_columns = 5;
  auto ctx = MakeCUDACtx(0);
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    auto x_device = thrust::device_vector<float>(x);
    for (auto num_bins : bin_sizes) {
      auto adapter = AdapterFromData(x_device, num_rows, num_columns);
      ValidateBatchedCuts(&ctx, adapter, num_bins, dmat.get());
    }
  }
}

TEST(HistUtil, AdapterDeviceSketchBatches) {
  int num_bins = 256;
  int num_rows = 5000;
  auto batch_sizes = {0, 100, 1500, 6000};
  int num_columns = 5;
  auto ctx = MakeCUDACtx(0);
  for (auto batch_size : batch_sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    auto x_device = thrust::device_vector<float>(x);
    auto adapter = AdapterFromData(x_device, num_rows, num_columns);
    ValidateBatchedCuts(&ctx, adapter, num_bins, dmat.get(), batch_size);
  }
}

namespace {
auto MakeData(Context const* ctx, std::size_t n_samples, bst_feature_t n_features) {
  dh::safe_cuda(hipSetDevice(ctx->Ordinal()));
  auto n = n_samples * n_features;
  std::vector<float> x;
  x.resize(n);

  std::iota(x.begin(), x.end(), 0);
  std::int32_t c{0};
  float missing = n_samples * n_features;
  for (std::size_t i = 0; i < x.size(); ++i) {
    if (i % 5 == 0) {
      x[i] = missing;
      c++;
    }
  }
  thrust::device_vector<float> d_x;
  d_x = x;

  auto n_invalids = n / 10 * 2 + 1;
  auto is_valid = data::IsValidFunctor{missing};
  return std::tuple{x, d_x, n_invalids, is_valid};
}

void TestGetColumnSize(std::size_t n_samples) {
  auto ctx = MakeCUDACtx(0);
  bst_feature_t n_features = 12;
  [[maybe_unused]] auto [x, d_x, n_invalids, is_valid] = MakeData(&ctx, n_samples, n_features);

  auto adapter = AdapterFromData(d_x, n_samples, n_features);
  auto batch = adapter.Value();

  auto batch_iter = dh::MakeTransformIterator<data::COOTuple>(
      thrust::make_counting_iterator(0llu),
      [=] __device__(std::size_t idx) { return batch.GetElement(idx); });

  dh::caching_device_vector<std::size_t> column_sizes_scan;
  column_sizes_scan.resize(n_features + 1);
  std::vector<std::size_t> h_column_size(column_sizes_scan.size());
  std::vector<std::size_t> h_column_size_1(column_sizes_scan.size());

  detail::LaunchGetColumnSizeKernel<decltype(batch_iter), true, true>(
      ctx.Device(), IterSpan{batch_iter, batch.Size()}, is_valid, dh::ToSpan(column_sizes_scan));
  thrust::copy(column_sizes_scan.begin(), column_sizes_scan.end(), h_column_size.begin());

  detail::LaunchGetColumnSizeKernel<decltype(batch_iter), true, false>(
      ctx.Device(), IterSpan{batch_iter, batch.Size()}, is_valid, dh::ToSpan(column_sizes_scan));
  thrust::copy(column_sizes_scan.begin(), column_sizes_scan.end(), h_column_size_1.begin());
  ASSERT_EQ(h_column_size, h_column_size_1);

  detail::LaunchGetColumnSizeKernel<decltype(batch_iter), false, true>(
      ctx.Device(), IterSpan{batch_iter, batch.Size()}, is_valid, dh::ToSpan(column_sizes_scan));
  thrust::copy(column_sizes_scan.begin(), column_sizes_scan.end(), h_column_size_1.begin());
  ASSERT_EQ(h_column_size, h_column_size_1);

  detail::LaunchGetColumnSizeKernel<decltype(batch_iter), false, false>(
      ctx.Device(), IterSpan{batch_iter, batch.Size()}, is_valid, dh::ToSpan(column_sizes_scan));
  thrust::copy(column_sizes_scan.begin(), column_sizes_scan.end(), h_column_size_1.begin());
  ASSERT_EQ(h_column_size, h_column_size_1);
}
}  // namespace

TEST(HistUtil, GetColumnSize) {
  bst_row_t n_samples = 4096;
  TestGetColumnSize(n_samples);
}

// Check sketching from adapter or DMatrix results in the same answer
// Consistency here is useful for testing and user experience
TEST(HistUtil, SketchingEquivalent) {
  auto ctx = MakeCUDACtx(0);
  auto bin_sizes = {2, 16, 256, 512};
  auto sizes = {100, 1000, 1500};
  int num_columns = 5;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    for (auto num_bins : bin_sizes) {
      auto dmat_cuts = DeviceSketch(&ctx, dmat.get(), num_bins);
      auto x_device = thrust::device_vector<float>(x);
      auto adapter = AdapterFromData(x_device, num_rows, num_columns);
      common::HistogramCuts adapter_cuts = MakeUnweightedCutsForTest(
          &ctx, adapter, num_bins, std::numeric_limits<float>::quiet_NaN());
      EXPECT_EQ(dmat_cuts.Values(), adapter_cuts.Values());
      EXPECT_EQ(dmat_cuts.Ptrs(), adapter_cuts.Ptrs());
      EXPECT_EQ(dmat_cuts.MinValues(), adapter_cuts.MinValues());

      ValidateBatchedCuts(&ctx, adapter, num_bins, dmat.get());
    }
  }
}

TEST(HistUtil, DeviceSketchFromGroupWeights) {
  auto ctx = MakeCUDACtx(0);
  size_t constexpr kRows = 3000, kCols = 200, kBins = 256;
  size_t constexpr kGroups = 10;
  auto m = RandomDataGenerator{kRows, kCols, 0}.GenerateDMatrix();

  // sketch with group weight
  auto& h_weights = m->Info().weights_.HostVector();
  h_weights.resize(kGroups);
  std::fill(h_weights.begin(), h_weights.end(), 1.0f);
  std::vector<bst_group_t> groups(kGroups);
  for (size_t i = 0; i < kGroups; ++i) {
    groups[i] = kRows / kGroups;
  }
  m->SetInfo("group", groups.data(), DataType::kUInt32, kGroups);
  HistogramCuts weighted_cuts = DeviceSketch(&ctx, m.get(), kBins, 0);

  // sketch with no weight
  h_weights.clear();
  HistogramCuts cuts = DeviceSketch(&ctx, m.get(), kBins, 0);

  ASSERT_EQ(cuts.Values().size(), weighted_cuts.Values().size());
  ASSERT_EQ(cuts.MinValues().size(), weighted_cuts.MinValues().size());
  ASSERT_EQ(cuts.Ptrs().size(), weighted_cuts.Ptrs().size());

  for (size_t i = 0; i < cuts.Values().size(); ++i) {
    EXPECT_EQ(cuts.Values()[i], weighted_cuts.Values()[i]) << "i:"<< i;
  }
  for (size_t i = 0; i < cuts.MinValues().size(); ++i) {
    ASSERT_EQ(cuts.MinValues()[i], weighted_cuts.MinValues()[i]);
  }
  for (size_t i = 0; i < cuts.Ptrs().size(); ++i) {
    ASSERT_EQ(cuts.Ptrs().at(i), weighted_cuts.Ptrs().at(i));
  }
  ValidateCuts(weighted_cuts, m.get(), kBins);
}

void TestAdapterSketchFromWeights(bool with_group) {
  size_t constexpr kRows = 300, kCols = 20, kBins = 256;
  size_t constexpr kGroups = 10;
  HostDeviceVector<float> storage;
  std::string m = RandomDataGenerator{kRows, kCols, 0}
                      .Device(DeviceOrd::CUDA(0))
                      .GenerateArrayInterface(&storage);
  MetaInfo info;
  auto ctx = MakeCUDACtx(0);
  auto& h_weights = info.weights_.HostVector();
  if (with_group) {
    h_weights.resize(kGroups);
  } else {
    h_weights.resize(kRows);
  }
  std::fill(h_weights.begin(), h_weights.end(), 1.0f);

  std::vector<bst_group_t> groups(kGroups);
  if (with_group) {
    for (size_t i = 0; i < kGroups; ++i) {
      groups[i] = kRows / kGroups;
    }
    info.SetInfo(ctx, "group", groups.data(), DataType::kUInt32, kGroups);
  }

  info.weights_.SetDevice(DeviceOrd::CUDA(0));
  info.num_row_ = kRows;
  info.num_col_ = kCols;

  data::CupyAdapter adapter(m);
  auto const& batch = adapter.Value();
  HostDeviceVector<FeatureType> ft;
  SketchContainer sketch_container(ft, kBins, kCols, kRows, DeviceOrd::CUDA(0));
  AdapterDeviceSketch(adapter.Value(), kBins, info, std::numeric_limits<float>::quiet_NaN(),
                      &sketch_container);

  common::HistogramCuts cuts;
  sketch_container.MakeCuts(&ctx, &cuts, info.IsColumnSplit());

  auto dmat = GetDMatrixFromData(storage.HostVector(), kRows, kCols);
  if (with_group) {
    dmat->Info().SetInfo(ctx, "group", groups.data(), DataType::kUInt32, kGroups);
  }

  dmat->Info().SetInfo(ctx, "weight", h_weights.data(), DataType::kFloat32, h_weights.size());
  dmat->Info().num_col_ = kCols;
  dmat->Info().num_row_ = kRows;
  ASSERT_EQ(cuts.Ptrs().size(), kCols + 1);
  ValidateCuts(cuts, dmat.get(), kBins);

  if (with_group) {
    dmat->Info().weights_ = decltype(dmat->Info().weights_)();  // remove weight
    HistogramCuts non_weighted = DeviceSketch(&ctx, dmat.get(), kBins, 0);
    for (size_t i = 0; i < cuts.Values().size(); ++i) {
      ASSERT_EQ(cuts.Values()[i], non_weighted.Values()[i]);
    }
    for (size_t i = 0; i < cuts.MinValues().size(); ++i) {
      ASSERT_EQ(cuts.MinValues()[i], non_weighted.MinValues()[i]);
    }
    for (size_t i = 0; i < cuts.Ptrs().size(); ++i) {
      ASSERT_EQ(cuts.Ptrs().at(i), non_weighted.Ptrs().at(i));
    }
  }

  if (with_group) {
    common::HistogramCuts weighted;
    auto& h_weights = info.weights_.HostVector();
    h_weights.resize(kGroups);
    // Generate different weight.
    for (size_t i = 0; i < h_weights.size(); ++i) {
      // FIXME(jiamingy): Some entries generated GPU test cannot pass the validate cuts if
      // we use more diverse weights, partially caused by
      // https://github.com/dmlc/xgboost/issues/7946
      h_weights[i] = (i % 2 == 0 ? 1 : 2) / static_cast<float>(kGroups);
    }
    SketchContainer sketch_container{ft, kBins, kCols, kRows, DeviceOrd::CUDA(0)};
    AdapterDeviceSketch(adapter.Value(), kBins, info, std::numeric_limits<float>::quiet_NaN(),
                        &sketch_container);
    sketch_container.MakeCuts(&ctx, &weighted, info.IsColumnSplit());
    ValidateCuts(weighted, dmat.get(), kBins);
  }
}

TEST(HistUtil, AdapterSketchFromWeights) {
  TestAdapterSketchFromWeights(false);
  TestAdapterSketchFromWeights(true);
}

namespace {
class DeviceSketchWithHessianTest
    : public ::testing::TestWithParam<std::tuple<bool, bst_row_t, bst_bin_t>> {
  bst_feature_t n_features_ = 5;
  bst_group_t n_groups_{3};

  auto GenerateHessian(Context const* ctx, bst_row_t n_samples) const {
    HostDeviceVector<float> hessian;
    auto& h_hess = hessian.HostVector();
    h_hess = GenerateRandomWeights(n_samples);
    std::mt19937 rng(0);
    std::shuffle(h_hess.begin(), h_hess.end(), rng);
    hessian.SetDevice(ctx->Device());
    return hessian;
  }

  void CheckReg(Context const* ctx, std::shared_ptr<DMatrix> p_fmat, bst_bin_t n_bins,
                HostDeviceVector<float> const& hessian, std::vector<float> const& w,
                std::size_t n_elements) const {
    auto const& h_hess = hessian.ConstHostVector();
    {
      auto& h_weight = p_fmat->Info().weights_.HostVector();
      h_weight = w;
    }

    HistogramCuts cuts_hess =
        DeviceSketchWithHessian(ctx, p_fmat.get(), n_bins, hessian.ConstDeviceSpan(), n_elements);
    ValidateCuts(cuts_hess, p_fmat.get(), n_bins);

    // merge hessian
    {
      auto& h_weight = p_fmat->Info().weights_.HostVector();
      ASSERT_EQ(h_weight.size(), h_hess.size());
      for (std::size_t i = 0; i < h_weight.size(); ++i) {
        h_weight[i] = w[i] * h_hess[i];
      }
    }

    HistogramCuts cuts_wh = DeviceSketch(ctx, p_fmat.get(), n_bins, n_elements);
    ValidateCuts(cuts_wh, p_fmat.get(), n_bins);
    ASSERT_EQ(cuts_hess.Values().size(), cuts_wh.Values().size());
    for (std::size_t i = 0; i < cuts_hess.Values().size(); ++i) {
      ASSERT_NEAR(cuts_wh.Values()[i], cuts_hess.Values()[i], kRtEps);
    }

    p_fmat->Info().weights_.HostVector() = w;
  }

 protected:
  Context ctx_ = MakeCUDACtx(0);

  void TestLTR(Context const* ctx, bst_row_t n_samples, bst_bin_t n_bins,
               std::size_t n_elements) const {
    auto x = GenerateRandom(n_samples, n_features_);

    std::vector<bst_group_t> gptr;
    gptr.resize(n_groups_ + 1, 0);
    gptr[1] = n_samples / n_groups_;
    gptr[2] = n_samples / n_groups_ + gptr[1];
    gptr.back() = n_samples;

    auto hessian = this->GenerateHessian(ctx, n_samples);
    auto const& h_hess = hessian.ConstHostVector();
    auto p_fmat = GetDMatrixFromData(x, n_samples, n_features_);
    p_fmat->Info().group_ptr_ = gptr;

    // test with constant group weight
    std::vector<float> w(n_groups_, 1.0f);
    p_fmat->Info().weights_.HostVector() = w;
    HistogramCuts cuts_hess =
        DeviceSketchWithHessian(ctx, p_fmat.get(), n_bins, hessian.ConstDeviceSpan(), n_elements);
    // make validation easier by converting it into sample weight.
    p_fmat->Info().weights_.HostVector() = h_hess;
    p_fmat->Info().group_ptr_.clear();
    ValidateCuts(cuts_hess, p_fmat.get(), n_bins);
    // restore ltr properties
    p_fmat->Info().weights_.HostVector() = w;
    p_fmat->Info().group_ptr_ = gptr;

    // test with random group weight
    w = GenerateRandomWeights(n_groups_);
    p_fmat->Info().weights_.HostVector() = w;
    cuts_hess =
        DeviceSketchWithHessian(ctx, p_fmat.get(), n_bins, hessian.ConstDeviceSpan(), n_elements);
    // make validation easier by converting it into sample weight.
    p_fmat->Info().weights_.HostVector() = h_hess;
    p_fmat->Info().group_ptr_.clear();
    ValidateCuts(cuts_hess, p_fmat.get(), n_bins);

    // merge hessian with sample weight
    p_fmat->Info().weights_.Resize(n_samples);
    p_fmat->Info().group_ptr_.clear();
    for (std::size_t i = 0; i < h_hess.size(); ++i) {
      auto gidx = dh::SegmentId(Span{gptr.data(), gptr.size()}, i);
      p_fmat->Info().weights_.HostVector()[i] = w[gidx] * h_hess[i];
    }
    auto cuts = DeviceSketch(ctx, p_fmat.get(), n_bins, n_elements);
    ValidateCuts(cuts, p_fmat.get(), n_bins);
    ASSERT_EQ(cuts.Values().size(), cuts_hess.Values().size());
    for (std::size_t i = 0; i < cuts.Values().size(); ++i) {
      EXPECT_NEAR(cuts.Values()[i], cuts_hess.Values()[i], 1e-4f);
    }
  }

  void TestRegression(Context const* ctx, bst_row_t n_samples, bst_bin_t n_bins,
                      std::size_t n_elements) const {
    auto x = GenerateRandom(n_samples, n_features_);
    auto p_fmat = GetDMatrixFromData(x, n_samples, n_features_);
    std::vector<float> w = GenerateRandomWeights(n_samples);

    auto hessian = this->GenerateHessian(ctx, n_samples);

    this->CheckReg(ctx, p_fmat, n_bins, hessian, w, n_elements);
  }
};

auto MakeParamsForTest() {
  std::vector<bst_row_t> sizes = {1, 2, 256, 512, 1000, 1500};
  std::vector<bst_bin_t> bin_sizes = {2, 16, 256, 512};
  std::vector<std::tuple<bool, bst_row_t, bst_bin_t>> configs;
  for (auto n_samples : sizes) {
    for (auto n_bins : bin_sizes) {
      configs.emplace_back(true, n_samples, n_bins);
      configs.emplace_back(false, n_samples, n_bins);
    }
  }
  return configs;
}
}  // namespace

TEST_P(DeviceSketchWithHessianTest, DeviceSketchWithHessian) {
  auto param = GetParam();
  auto n_samples = std::get<1>(param);
  auto n_bins = std::get<2>(param);
  if (std::get<0>(param)) {
    this->TestLTR(&ctx_, n_samples, n_bins, 0);
    this->TestLTR(&ctx_, n_samples, n_bins, 512);
  } else {
    this->TestRegression(&ctx_, n_samples, n_bins, 0);
    this->TestRegression(&ctx_, n_samples, n_bins, 512);
  }
}

INSTANTIATE_TEST_SUITE_P(
    HistUtil, DeviceSketchWithHessianTest, ::testing::ValuesIn(MakeParamsForTest()),
    [](::testing::TestParamInfo<DeviceSketchWithHessianTest::ParamType> const& info) {
      auto task = std::get<0>(info.param) ? "ltr" : "reg";
      auto n_samples = std::to_string(std::get<1>(info.param));
      auto n_bins = std::to_string(std::get<2>(info.param));
      return std::string{task} + "_" + n_samples + "_" + n_bins;
    });
}  // namespace xgboost::common
