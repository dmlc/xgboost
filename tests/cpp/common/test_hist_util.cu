#include <dmlc/filesystem.h>
#include <gtest/gtest.h>

#include <algorithm>
#include <cmath>


#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>

#include "xgboost/c_api.h"

#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/hist_util.h"

#include "../helpers.h"
#include <xgboost/data.h>
#include "../../../src/data/device_adapter.cuh"
#include "../data/test_array_interface.h"
#include "../../../src/common/math.h"
#include "../../../src/data/simple_dmatrix.h"
#include "test_hist_util.h"

namespace xgboost {
namespace common {

template <typename AdapterT>
HistogramCuts GetHostCuts(AdapterT *adapter, int num_bins, float missing) {
  HistogramCuts cuts;
  DenseCuts builder(&cuts);
  data::SimpleDMatrix dmat(adapter, missing, 1);
  builder.Build(&dmat, num_bins);
  return cuts;
}
TEST(hist_util, DeviceSketch) {
  int num_rows = 5;
  int num_columns = 1;
  int num_bins = 4;
  std::vector<float> x = {1.0, 2.0, 3.0, 4.0, 5.0};
  auto dmat = GetDMatrixFromData(x, num_rows, num_columns);

  auto device_cuts = DeviceSketch(0, dmat.get(), num_bins, 0);
  HistogramCuts host_cuts;
  DenseCuts builder(&host_cuts);
  builder.Build(dmat.get(), num_bins);

  EXPECT_EQ(device_cuts.Values(), host_cuts.Values());
  EXPECT_EQ(device_cuts.Ptrs(), host_cuts.Ptrs());
  EXPECT_EQ(device_cuts.MinValues(), host_cuts.MinValues());
}

TEST(hist_util, DeviceSketchDeterminism) {
  int num_rows = 500;
  int num_columns = 5;
  int num_bins = 256;
  auto x = GenerateRandom(num_rows, num_columns);
  auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
  auto reference_sketch = DeviceSketch(0, dmat.get(), num_bins);
  size_t constexpr kRounds{ 100 };
  for (size_t r = 0; r < kRounds; ++r) {
    auto new_sketch = DeviceSketch(0, dmat.get(), num_bins);
    ASSERT_EQ(reference_sketch.Values(), new_sketch.Values());
    ASSERT_EQ(reference_sketch.MinValues(), new_sketch.MinValues());
  }
}

 TEST(hist_util, DeviceSketchCategorical) {
  int categorical_sizes[] = {2, 6, 8, 12};
  int num_bins = 256;
  int sizes[] = {25, 100, 1000};
  for (auto n : sizes) {
    for (auto num_categories : categorical_sizes) {
      auto x = GenerateRandomCategoricalSingleColumn(n, num_categories);
      auto dmat = GetDMatrixFromData(x, n, 1);
      auto cuts = DeviceSketch(0, dmat.get(), num_bins, 0);
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}

TEST(hist_util, DeviceSketchMultipleColumns) {
  int bin_sizes[] = {2, 16, 256, 512};
  int sizes[] = {100, 1000, 1500};
  int num_columns = 5;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    for (auto num_bins : bin_sizes) {
      auto cuts = DeviceSketch(0, dmat.get(), num_bins, 0);
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }

}

TEST(hist_util, DeviceSketchMultipleColumnsWeights) {
  int bin_sizes[] = {2, 16, 256, 512};
  int sizes[] = {100, 1000, 1500};
  int num_columns = 5;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    dmat->Info().weights_.HostVector() = GenerateRandomWeights(num_rows);
    for (auto num_bins : bin_sizes) {
      auto cuts = DeviceSketch(0, dmat.get(), num_bins, 0);
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}

TEST(hist_util, DeviceSketchBatches) {
  int num_bins = 256;
  int num_rows = 5000;
  int batch_sizes[] = {0, 100, 1500, 6000};
  int num_columns = 5;
  for (auto batch_size : batch_sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    auto cuts = DeviceSketch(0, dmat.get(), num_bins, batch_size);
    ValidateCuts(cuts, dmat.get(), num_bins);
  }
}

TEST(hist_util, DeviceSketchMultipleColumnsExternal) {
  int bin_sizes[] = {2, 16, 256, 512};
  int sizes[] = {100, 1000, 1500};
  int num_columns =5;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    dmlc::TemporaryDirectory temp;
    auto dmat =
        GetExternalMemoryDMatrixFromData(x, num_rows, num_columns, 100, temp);
    for (auto num_bins : bin_sizes) {
      auto cuts = DeviceSketch(0, dmat.get(), num_bins, 0);
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}

TEST(hist_util, AdapterDeviceSketch)
{
  int rows = 5;
  int cols = 1;
  int num_bins = 4;
  float missing =  - 1.0;
  thrust::device_vector< float> data(rows*cols);
  auto json_array_interface = Generate2dArrayInterface(rows, cols, "<f4", &data);
  data = std::vector<float >{ 1.0,2.0,3.0,4.0,5.0 };
  std::stringstream ss;
  Json::Dump(json_array_interface, &ss);
  std::string str = ss.str();
  data::CupyAdapter adapter(str);

  auto device_cuts = AdapterDeviceSketch(&adapter, num_bins, missing);
  auto host_cuts = GetHostCuts(&adapter, num_bins, missing);

  EXPECT_EQ(device_cuts.Values(), host_cuts.Values());
  EXPECT_EQ(device_cuts.Ptrs(), host_cuts.Ptrs());
  EXPECT_EQ(device_cuts.MinValues(), host_cuts.MinValues());
}

 TEST(hist_util, AdapterDeviceSketchCategorical) {
  int categorical_sizes[] = {2, 6, 8, 12};
  int num_bins = 256;
  int sizes[] = {25, 100, 1000};
  for (auto n : sizes) {
    for (auto num_categories : categorical_sizes) {
      auto x = GenerateRandomCategoricalSingleColumn(n, num_categories);
      auto dmat = GetDMatrixFromData(x, n, 1);
      auto x_device = thrust::device_vector<float>(x);
      auto adapter = AdapterFromData(x_device, n, 1);
      auto cuts = AdapterDeviceSketch(&adapter, num_bins,
                                      std::numeric_limits<float>::quiet_NaN());
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}

TEST(hist_util, AdapterDeviceSketchMultipleColumns) {
  int bin_sizes[] = {2, 16, 256, 512};
  int sizes[] = {100, 1000, 1500};
  int num_columns = 5;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    auto x_device = thrust::device_vector<float>(x);
    for (auto num_bins : bin_sizes) {
      auto adapter = AdapterFromData(x_device, num_rows, num_columns);
      auto cuts = AdapterDeviceSketch(&adapter, num_bins,
                                      std::numeric_limits<float>::quiet_NaN());
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}
TEST(hist_util, AdapterDeviceSketchBatches) {
  int num_bins = 256;
  int num_rows = 5000;
  int batch_sizes[] = {0, 100, 1500, 6000};
  int num_columns = 5;
  for (auto batch_size : batch_sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    auto x_device = thrust::device_vector<float>(x);
    auto adapter = AdapterFromData(x_device, num_rows, num_columns);
    auto cuts = AdapterDeviceSketch(&adapter, num_bins,
                                    std::numeric_limits<float>::quiet_NaN(),
                                    batch_size);
    ValidateCuts(cuts, dmat.get(), num_bins);
  }
}
}  // namespace common
}  // namespace xgboost
