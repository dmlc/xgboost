#include "hip/hip_runtime.h"
#include <dmlc/filesystem.h>
#include <gtest/gtest.h>

#include <algorithm>
#include <cmath>


#include <thrust/device_vector.h>

#include "xgboost/c_api.h"

#include "../../../src/common/device_helpers.cuh"
#include "../../../src/common/hist_util.h"

#include "../helpers.h"
#include <xgboost/data.h>
#include "../../../src/data/device_adapter.cuh"
#include "../data/test_array_interface.h"
#include "../../../src/common/math.h"
#include "../../../src/data/simple_dmatrix.h"
#include "test_hist_util.h"
#include "../../../include/xgboost/logging.h"

namespace xgboost {
namespace common {

template <typename AdapterT>
HistogramCuts GetHostCuts(AdapterT *adapter, int num_bins, float missing) {
  HistogramCuts cuts;
  DenseCuts builder(&cuts);
  data::SimpleDMatrix dmat(adapter, missing, 1);
  builder.Build(&dmat, num_bins);
  return cuts;
}
TEST(hist_util, DeviceSketch) {
  int num_rows = 5;
  int num_columns = 1;
  int num_bins = 4;
  std::vector<float> x = {1.0, 2.0, 3.0, 4.0, 5.0};
  auto dmat = GetDMatrixFromData(x, num_rows, num_columns);

  auto device_cuts = DeviceSketch(0, dmat.get(), num_bins);
  HistogramCuts host_cuts;
  DenseCuts builder(&host_cuts);
  builder.Build(dmat.get(), num_bins);

  EXPECT_EQ(device_cuts.Values(), host_cuts.Values());
  EXPECT_EQ(device_cuts.Ptrs(), host_cuts.Ptrs());
  EXPECT_EQ(device_cuts.MinValues(), host_cuts.MinValues());
}

// Duplicate this function from hist_util.cu so we don't have to expose it in
// header
size_t RequiredSampleCutsTest(int max_bins, size_t num_rows) {
  constexpr int kFactor = 8;
  double eps = 1.0 / (kFactor * max_bins);
  size_t dummy_nlevel;
  size_t num_cuts;
  WQuantileSketch<bst_float, bst_float>::LimitSizeLevel(
    num_rows, eps, &dummy_nlevel, &num_cuts);
  return std::min(num_cuts, num_rows);
}

TEST(hist_util, DeviceSketchMemory) {
  int num_columns = 100;
  int num_rows = 1000;
  int num_bins = 256;
  auto x = GenerateRandom(num_rows, num_columns);
  auto dmat = GetDMatrixFromData(x, num_rows, num_columns);

  dh::GlobalMemoryLogger().Clear();
  ConsoleLogger::Configure({{"verbosity", "3"}});
  auto device_cuts = DeviceSketch(0, dmat.get(), num_bins);
  ConsoleLogger::Configure({{"verbosity", "0"}});

  size_t bytes_num_elements = num_rows * num_columns*sizeof(Entry);
  size_t bytes_cuts = RequiredSampleCutsTest(num_bins, num_rows) * num_columns *
                      sizeof(DenseCuts::WQSketch::Entry);
  size_t bytes_constant = 1000;
  EXPECT_LE(dh::GlobalMemoryLogger().PeakMemory(),
            bytes_num_elements + bytes_cuts + bytes_constant);
}

TEST(hist_util, DeviceSketchMemoryWeights) {
  int num_columns = 100;
  int num_rows = 1000;
  int num_bins = 256;
  auto x = GenerateRandom(num_rows, num_columns);
  auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
  dmat->Info().weights_.HostVector() = GenerateRandomWeights(num_rows);

  dh::GlobalMemoryLogger().Clear();
  ConsoleLogger::Configure({{"verbosity", "3"}});
  auto device_cuts = DeviceSketch(0, dmat.get(), num_bins);
  ConsoleLogger::Configure({{"verbosity", "0"}});

  size_t bytes_num_elements =
      num_rows * num_columns * (sizeof(Entry) + sizeof(float));
  size_t bytes_cuts = RequiredSampleCutsTest(num_bins, num_rows) * num_columns *
                      sizeof(DenseCuts::WQSketch::Entry);
  EXPECT_LE(dh::GlobalMemoryLogger().PeakMemory(),
            size_t((bytes_num_elements + bytes_cuts) * 1.05));
}

TEST(hist_util, DeviceSketchDeterminism) {
  int num_rows = 500;
  int num_columns = 5;
  int num_bins = 256;
  auto x = GenerateRandom(num_rows, num_columns);
  auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
  auto reference_sketch = DeviceSketch(0, dmat.get(), num_bins);
  size_t constexpr kRounds{ 100 };
  for (size_t r = 0; r < kRounds; ++r) {
    auto new_sketch = DeviceSketch(0, dmat.get(), num_bins);
    ASSERT_EQ(reference_sketch.Values(), new_sketch.Values());
    ASSERT_EQ(reference_sketch.MinValues(), new_sketch.MinValues());
  }
}

 TEST(hist_util, DeviceSketchCategorical) {
  int categorical_sizes[] = {2, 6, 8, 12};
  int num_bins = 256;
  int sizes[] = {25, 100, 1000};
  for (auto n : sizes) {
    for (auto num_categories : categorical_sizes) {
      auto x = GenerateRandomCategoricalSingleColumn(n, num_categories);
      auto dmat = GetDMatrixFromData(x, n, 1);
      auto cuts = DeviceSketch(0, dmat.get(), num_bins);
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}

TEST(hist_util, DeviceSketchMultipleColumns) {
  int bin_sizes[] = {2, 16, 256, 512};
  int sizes[] = {100, 1000, 1500};
  int num_columns = 5;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    for (auto num_bins : bin_sizes) {
      auto cuts = DeviceSketch(0, dmat.get(), num_bins);
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }

}

TEST(hist_util, DeviceSketchMultipleColumnsWeights) {
  int bin_sizes[] = {2, 16, 256, 512};
  int sizes[] = {100, 1000, 1500};
  int num_columns = 5;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    dmat->Info().weights_.HostVector() = GenerateRandomWeights(num_rows);
    for (auto num_bins : bin_sizes) {
      auto cuts = DeviceSketch(0, dmat.get(), num_bins);
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}

TEST(hist_util, DeviceSketchBatches) {
  int num_bins = 256;
  int num_rows = 5000;
  int batch_sizes[] = {0, 100, 1500, 6000};
  int num_columns = 5;
  for (auto batch_size : batch_sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    auto cuts = DeviceSketch(0, dmat.get(), num_bins, batch_size);
    ValidateCuts(cuts, dmat.get(), num_bins);
  }
}

TEST(hist_util, DeviceSketchMultipleColumnsExternal) {
  int bin_sizes[] = {2, 16, 256, 512};
  int sizes[] = {100, 1000, 1500};
  int num_columns =5;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    dmlc::TemporaryDirectory temp;
    auto dmat =
        GetExternalMemoryDMatrixFromData(x, num_rows, num_columns, 100, temp);
    for (auto num_bins : bin_sizes) {
      auto cuts = DeviceSketch(0, dmat.get(), num_bins);
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}

TEST(hist_util, AdapterDeviceSketch)
{
  int rows = 5;
  int cols = 1;
  int num_bins = 4;
  float missing =  - 1.0;
  thrust::device_vector< float> data(rows*cols);
  auto json_array_interface = Generate2dArrayInterface(rows, cols, "<f4", &data);
  data = std::vector<float >{ 1.0,2.0,3.0,4.0,5.0 };
  std::stringstream ss;
  Json::Dump(json_array_interface, &ss);
  std::string str = ss.str();
  data::CupyAdapter adapter(str);

  auto device_cuts = AdapterDeviceSketch(&adapter, num_bins, missing);
  auto host_cuts = GetHostCuts(&adapter, num_bins, missing);

  EXPECT_EQ(device_cuts.Values(), host_cuts.Values());
  EXPECT_EQ(device_cuts.Ptrs(), host_cuts.Ptrs());
  EXPECT_EQ(device_cuts.MinValues(), host_cuts.MinValues());
}

TEST(hist_util, AdapterDeviceSketchMemory) {
  int num_columns = 100;
  int num_rows = 1000;
  int num_bins = 256;
  auto x = GenerateRandom(num_rows, num_columns);
  auto x_device = thrust::device_vector<float>(x);
  auto adapter = AdapterFromData(x_device, num_rows, num_columns);

  dh::GlobalMemoryLogger().Clear();
  ConsoleLogger::Configure({{"verbosity", "3"}});
  auto cuts = AdapterDeviceSketch(&adapter, num_bins,
                                  std::numeric_limits<float>::quiet_NaN());
  ConsoleLogger::Configure({{"verbosity", "0"}});

  size_t bytes_num_elements = num_rows * num_columns * sizeof(Entry);
  size_t bytes_num_columns = (num_columns + 1) * sizeof(size_t);
  size_t bytes_cuts = RequiredSampleCutsTest(num_bins, num_rows) * num_columns *
                      sizeof(DenseCuts::WQSketch::Entry);
  size_t bytes_constant = 1000;
  EXPECT_LE(dh::GlobalMemoryLogger().PeakMemory(),
      bytes_num_elements + bytes_cuts + bytes_num_columns + bytes_constant);
}

 TEST(hist_util, AdapterDeviceSketchCategorical) {
  int categorical_sizes[] = {2, 6, 8, 12};
  int num_bins = 256;
  int sizes[] = {25, 100, 1000};
  for (auto n : sizes) {
    for (auto num_categories : categorical_sizes) {
      auto x = GenerateRandomCategoricalSingleColumn(n, num_categories);
      auto dmat = GetDMatrixFromData(x, n, 1);
      auto x_device = thrust::device_vector<float>(x);
      auto adapter = AdapterFromData(x_device, n, 1);
      auto cuts = AdapterDeviceSketch(&adapter, num_bins,
                                      std::numeric_limits<float>::quiet_NaN());
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}

TEST(hist_util, AdapterDeviceSketchMultipleColumns) {
  int bin_sizes[] = {2, 16, 256, 512};
  int sizes[] = {100, 1000, 1500};
  int num_columns = 5;
  for (auto num_rows : sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    auto x_device = thrust::device_vector<float>(x);
    for (auto num_bins : bin_sizes) {
      auto adapter = AdapterFromData(x_device, num_rows, num_columns);
      auto cuts = AdapterDeviceSketch(&adapter, num_bins,
                                      std::numeric_limits<float>::quiet_NaN());
      ValidateCuts(cuts, dmat.get(), num_bins);
    }
  }
}
TEST(hist_util, AdapterDeviceSketchBatches) {
  int num_bins = 256;
  int num_rows = 5000;
  int batch_sizes[] = {0, 100, 1500, 6000};
  int num_columns = 5;
  for (auto batch_size : batch_sizes) {
    auto x = GenerateRandom(num_rows, num_columns);
    auto dmat = GetDMatrixFromData(x, num_rows, num_columns);
    auto x_device = thrust::device_vector<float>(x);
    auto adapter = AdapterFromData(x_device, num_rows, num_columns);
    auto cuts = AdapterDeviceSketch(&adapter, num_bins,
                                    std::numeric_limits<float>::quiet_NaN(),
                                    batch_size);
    ValidateCuts(cuts, dmat.get(), num_bins);
  }
}
}  // namespace common
}  // namespace xgboost
