/**
 * Copyright 2023 by XGBoost Contributors
 */
#include <gtest/gtest.h>
#include <xgboost/context.h>                     // for Context

#include <cstdint>                               // for uint32_t
#include <vector>                                // for vector

#include "../../../src/common/cuda_context.cuh"  // for HIPContext
#include "../../../src/objective/lambdarank_obj.cuh"
#include "test_lambdarank_obj.h"

namespace xgboost::obj {
TEST(LambdaRank, GPUNDCGJsonIO) {
  auto ctx = MakeCUDACtx(0);
  TestNDCGJsonIO(&ctx);
}

TEST(LambdaRank, GPUMAPStat) {
  auto ctx = MakeCUDACtx(0);
  TestMAPStat(&ctx);
}

TEST(LambdaRank, GPUNDCGGPair) {
  auto ctx = MakeCUDACtx(0);
  TestNDCGGPair(&ctx);
}

void TestGPUMakePair() {
  auto ctx = MakeCUDACtx(0);

  MetaInfo info;
  HostDeviceVector<float> predt;
  InitMakePairTest(&ctx, &info, &predt);

  ltr::LambdaRankParam param;

  auto make_args = [&](std::shared_ptr<ltr::RankingCache> p_cache, auto rank_idx,
                       common::Span<std::size_t const> y_sorted_idx) {
    linalg::Vector<double> dummy;
    auto d = dummy.View(ctx.Device());
    linalg::Vector<GradientPair> dgpair;
    auto dg = dgpair.View(ctx.Device());
    cuda_impl::KernelInputs args{
        d,
        d,
        d,
        d,
        p_cache->DataGroupPtr(&ctx),
        p_cache->CUDAThreadsGroupPtr(),
        rank_idx,
        info.labels.View(ctx.Device()),
        predt.ConstDeviceSpan(),
        linalg::MatrixView<GradientPair>{common::Span<GradientPair>{}, {0}, DeviceOrd::CUDA(0)},
        dg,
        nullptr,
        y_sorted_idx,
        0};
    return args;
  };

  {
    param.UpdateAllowUnknown(Args{{"lambdarank_pair_method", "topk"}});
    auto p_cache = std::make_shared<ltr::NDCGCache>(&ctx, info, param);
    auto rank_idx = p_cache->SortedIdx(&ctx, predt.ConstDeviceSpan());

    ASSERT_EQ(p_cache->CUDAThreads(), 3568);

    auto args = make_args(p_cache, rank_idx, {});
    auto n_pairs = p_cache->Param().NumPair();
    auto make_pair = cuda_impl::MakePairsOp<true>{args};

    dh::LaunchN(p_cache->CUDAThreads(), ctx.CUDACtx()->Stream(),
                [=] XGBOOST_DEVICE(std::size_t idx) {
                  auto [i, j] = make_pair(idx, 0);
                  SPAN_CHECK(j > i);
                  SPAN_CHECK(i < n_pairs);
                });
  }
  {
    param.UpdateAllowUnknown(Args{{"lambdarank_pair_method", "mean"}});
    auto p_cache = std::make_shared<ltr::NDCGCache>(&ctx, info, param);
    auto rank_idx = p_cache->SortedIdx(&ctx, predt.ConstDeviceSpan());
    auto y_sorted_idx = cuda_impl::SortY(&ctx, info, rank_idx, p_cache);

    ASSERT_FALSE(param.HasTruncation());
    ASSERT_EQ(p_cache->CUDAThreads(), info.num_row_ * param.NumPair());

    auto args = make_args(p_cache, rank_idx, y_sorted_idx);
    auto make_pair = cuda_impl::MakePairsOp<false>{args};
    auto n_pairs = p_cache->Param().NumPair();
    ASSERT_EQ(n_pairs, 1);

    dh::LaunchN(
        p_cache->CUDAThreads(), ctx.CUDACtx()->Stream(), [=] XGBOOST_DEVICE(std::size_t idx) {
          idx = 97;
          auto [i, j] = make_pair(idx, 0);
          // Not in the same bucket
          SPAN_CHECK(make_pair.args.labels(rank_idx[i]) != make_pair.args.labels(rank_idx[j]));
        });
  }
  {
    param.UpdateAllowUnknown(Args{{"lambdarank_num_pair_per_sample", "2"}});
    auto p_cache = std::make_shared<ltr::NDCGCache>(&ctx, info, param);
    auto rank_idx = p_cache->SortedIdx(&ctx, predt.ConstDeviceSpan());
    auto y_sorted_idx = cuda_impl::SortY(&ctx, info, rank_idx, p_cache);

    auto args = make_args(p_cache, rank_idx, y_sorted_idx);
    auto make_pair = cuda_impl::MakePairsOp<false>{args};

    dh::LaunchN(
        p_cache->CUDAThreads(), ctx.CUDACtx()->Stream(), [=] XGBOOST_DEVICE(std::size_t idx) {
          auto [i, j] = make_pair(idx, 0);
          // Not in the same bucket
          SPAN_CHECK(make_pair.args.labels(rank_idx[i]) != make_pair.args.labels(rank_idx[j]));
        });
    ASSERT_EQ(param.NumPair(), 2);
    ASSERT_EQ(p_cache->CUDAThreads(), info.num_row_ * param.NumPair());
  }
}

TEST(LambdaRank, GPUMakePair) { TestGPUMakePair(); }

TEST(LambdaRank, GPUUnbiasedNDCG) {
  auto ctx = MakeCUDACtx(0);
  TestUnbiasedNDCG(&ctx);
}

template <typename CountFunctor>
void RankItemCountImpl(std::vector<std::uint32_t> const &sorted_items, CountFunctor f,
                       std::uint32_t find_val, std::uint32_t exp_val) {
  EXPECT_NE(std::find(sorted_items.begin(), sorted_items.end(), find_val), sorted_items.end());
  EXPECT_EQ(f(&sorted_items[0], sorted_items.size(), find_val), exp_val);
}

TEST(LambdaRank, RankItemCountOnLeft) {
  // Items sorted descendingly
  std::vector<std::uint32_t> sorted_items{10, 10, 6, 4, 4, 4, 4, 1, 1, 1, 1, 1, 0};
  auto wrapper = [](auto const &...args) { return cuda_impl::CountNumItemsToTheLeftOf(args...); };
  RankItemCountImpl(sorted_items, wrapper, 10, static_cast<uint32_t>(0));
  RankItemCountImpl(sorted_items, wrapper, 6, static_cast<uint32_t>(2));
  RankItemCountImpl(sorted_items, wrapper, 4, static_cast<uint32_t>(3));
  RankItemCountImpl(sorted_items, wrapper, 1, static_cast<uint32_t>(7));
  RankItemCountImpl(sorted_items, wrapper, 0, static_cast<uint32_t>(12));
}

TEST(LambdaRank, RankItemCountOnRight) {
  // Items sorted descendingly
  std::vector<std::uint32_t> sorted_items{10, 10, 6, 4, 4, 4, 4, 1, 1, 1, 1, 1, 0};
  auto wrapper = [](auto const &...args) { return cuda_impl::CountNumItemsToTheRightOf(args...); };
  RankItemCountImpl(sorted_items, wrapper, 10, static_cast<uint32_t>(11));
  RankItemCountImpl(sorted_items, wrapper, 6, static_cast<uint32_t>(10));
  RankItemCountImpl(sorted_items, wrapper, 4, static_cast<uint32_t>(6));
  RankItemCountImpl(sorted_items, wrapper, 1, static_cast<uint32_t>(1));
  RankItemCountImpl(sorted_items, wrapper, 0, static_cast<uint32_t>(0));
}

TEST(LambdaRank, GPUMAPGPair) {
  auto ctx = MakeCUDACtx(0);
  TestMAPGPair(&ctx);
}
}  // namespace xgboost::obj
