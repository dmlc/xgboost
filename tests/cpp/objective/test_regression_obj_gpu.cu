/*!
 * Copyright 2017 XGBoost contributors
 */
#include <xgboost/objective.h>

#include "../helpers.h"

TEST(Objective, GPULinearRegressionGPair) {
  xgboost::ObjFunction * obj = xgboost::ObjFunction::Create("gpu:reg:linear");
  std::vector<std::pair<std::string, std::string> > args;
  obj->Configure(args);
  CheckObjFunction(obj,
                   {0, 0.1f, 0.9f,   1,    0,  0.1f, 0.9f,  1},
                   {0,   0,   0,   0,    1,    1,    1, 1},
                   {1,   1,   1,   1,    1,    1,    1, 1},
                   {0, 0.1f, 0.9f, 1.0f, -1.0f, -0.9f, -0.1f, 0},
                   {1,   1,   1,   1,    1,    1,    1, 1});

  ASSERT_NO_THROW(obj->DefaultEvalMetric());
}

TEST(Objective, GPULogisticRegressionGPair) {
  xgboost::ObjFunction * obj = xgboost::ObjFunction::Create("gpu:reg:logistic");
  std::vector<std::pair<std::string, std::string> > args;
  obj->Configure(args);
  CheckObjFunction(obj,
                   {   0,  0.1f,  0.9f,    1,    0,   0.1f,  0.9f,      1},
                   {   0,    0,    0,    0,    1,     1,     1,     1},
                   {   1,    1,    1,    1,    1,     1,     1,     1},
                   { 0.5f, 0.52f, 0.71f, 0.73f, -0.5f, -0.47f, -0.28f, -0.26f},
                   {0.25f, 0.24f, 0.20f, 0.19f, 0.25f,  0.24f,  0.20f,  0.19f});
}

TEST(Objective, GPULogisticRegressionBasic) {
  xgboost::ObjFunction * obj = xgboost::ObjFunction::Create("gpu:reg:logistic");
  std::vector<std::pair<std::string, std::string> > args;
  obj->Configure(args);

  // test label validation
  EXPECT_ANY_THROW(CheckObjFunction(obj, {0}, {10}, {1}, {0}, {0}))
    << "Expected error when label not in range [0,1f] for LogisticRegression";

  // test ProbToMargin
  EXPECT_NEAR(obj->ProbToMargin(0.1f), -2.197f, 0.01f);
  EXPECT_NEAR(obj->ProbToMargin(0.5f), 0, 0.01f);
  EXPECT_NEAR(obj->ProbToMargin(0.9f), 2.197f, 0.01f);
  EXPECT_ANY_THROW(obj->ProbToMargin(10))
    << "Expected error when base_score not in range [0,1f] for LogisticRegression";

  // test PredTransform
  std::vector<xgboost::bst_float> preds = {0, 0.1f, 0.5f, 0.9f, 1};
  std::vector<xgboost::bst_float> out_preds = {0.5f, 0.524f, 0.622f, 0.710f, 0.731f};
  obj->PredTransform(&preds);
  for (int i = 0; i < static_cast<int>(preds.size()); ++i) {
    EXPECT_NEAR(preds[i], out_preds[i], 0.01f);
  }
}

TEST(Objective, GPULogisticRawGPair) {
  xgboost::ObjFunction * obj = xgboost::ObjFunction::Create("gpu:binary:logitraw");
  std::vector<std::pair<std::string, std::string> > args;
  obj->Configure(args);
  CheckObjFunction(obj,
                   {   0,  0.1f,  0.9f,    1,    0,   0.1f,   0.9f,     1},
                   {   0,    0,    0,    0,    1,     1,     1,     1},
                   {   1,    1,    1,    1,    1,     1,     1,     1},
                   { 0.5f, 0.52f, 0.71f, 0.73f, -0.5f, -0.47f, -0.28f, -0.26f},
                   {0.25f, 0.24f, 0.20f, 0.19f, 0.25f,  0.24f,  0.20f,  0.19f});
}
